#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>

#define TAM_TABLE 255
#define TAM_TH 127

#define N_ELEM 50000000
// #define N_ELEM 100000000

#define INDIRECT

#define _RF_TABLE_1TREE_CONST
// #define _RF_TABLE_2TREE_CONST
// #define _RF_TABLE_3TREE_CONST
// #define _RF_TABLE_4TREE_CONST

#define WARP_SIZE 32

#define CHECK(call)                                                \
	{                                                              \
		const hipError_t error = call;                            \
		if (error != hipSuccess)                                  \
		{                                                          \
			fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
			fprintf(stderr, "code: %d, reason: %s\n", error,       \
					hipGetErrorString(error));                    \
		}                                                          \
	}

__constant__ int table[TAM_TABLE];
__constant__ float TH[TAM_TH];

#define _MOD 64

#if defined(_RF_TABLE_1TREE_CONST)
#define _1TREE
#define EXP_NAME "_RF_TABLE_1TREE_CONST"
#endif

#if defined(_RF_TABLE_2TREE_CONST)
#define _2TREE
#define EXP_NAME "_RF_TABLE_2TREE_CONST"
#endif

#if defined(_RF_TABLE_3TREE_CONST)
#define _3TREE
#define EXP_NAME "_RF_TABLE_3TREE_CONST"
#endif

#if defined(_RF_TABLE_4TREE_CONST)
#define _4TREE
#define EXP_NAME "_RF_TABLE_4TREE_CONST"
#endif
__global__ void RF_kernel(float *input, const int n, const int offset0, const int offset1, const int offset2, const int offset3, float *vR)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int next, left, right;
	float th;

#ifdef INDIRECT
	int idx;
	int v[4];

	v[1] = 12;
	v[2] = 128;
	v[3] = 1;
#else
	int v;
#endif
	if (i < n)
	{

#ifdef INDIRECT
		v[0] = input[i];
#else
		v = input[i];
#endif
		next = 0;
#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset0)) ? left : right;
#else
		next = (v > (th + offset0)) ? left : right;
#endif

		// if (i == 2)
		//	printf("idx=%d, left=%d , right=%d , th=%f , next=%d \n", idx, left, right, th, next);

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset0)) ? left : right;
#else
		next = (v > (th + offset0)) ? left : right;
#endif

		// if (i == 2)
		//	printf("idx=%d, left=%d , right=%d , th=%f , next=%d \n", idx, left, right, th, next);

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset0)) ? left : right;
#else
		next = (v > (th + offset0)) ? left : right;
#endif

		// if (i == 2)
		//	printf("idx=%d, left=%d , right=%d , th=%f , next=%d \n", idx, left, right, th, next);

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset0)) ? left : right;
#else
		next = (v > (th + offset0)) ? left : right;
#endif

		// if (i == 2)
		//	printf("idx=%d, left=%d , right=%d , th=%f , next=%d \n", idx, left, right, th, next);

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset0)) ? left : right;
#else
		next = (v > (th + offset0)) ? left : right;
#endif

		// if (i == 2)
		//	printf("idx=%d, left=%d , right=%d , th=%f , next=%d \n", idx, left, right, th, next);

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset0)) ? left : right;
#else
		next = (v > (th + offset0)) ? left : right;
#endif

		// if (i == 2)
		//	printf("idx=%d, left=%d , right=%d , th=%f , next=%d \n", idx, left, right, th, next);

		vR[i] = table[next] + offset0;

#if defined(_RF_TABLE_2TREE_CONST) || defined(_RF_TABLE_3TREE_CONST) || defined(_RF_TABLE_4TREE_CONST)
#ifdef INDIRECT
		v[0] = input[i];
#else
		v = input[i];
#endif
		next = 0;
#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset1)) ? left : right;
#else
		next = (v > (th + offset1)) ? left : right;
#endif

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset1)) ? left : right;
#else
		next = (v > (th + offset1)) ? left : right;
#endif

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset1)) ? left : right;
#else
		next = (v > (th + offset1)) ? left : right;
#endif

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset1)) ? left : right;
#else
		next = (v > (th + offset1)) ? left : right;
#endif

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset1)) ? left : right;
#else
		next = (v > (th + offset1)) ? left : right;
#endif

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset1)) ? left : right;
#else
		next = (v > (th + offset1)) ? left : right;
#endif

		vR[i] = table[next] + offset1;
#endif

#if defined(_RF_TABLE_3TREE_CONST) || defined(_RF_TABLE_4TREE_CONST)
#ifdef INDIRECT
		v[0] = input[i];
#else
		v = input[i];
#endif
		next = 0;
#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset2)) ? left : right;
#else
		next = (v > (th + offset2)) ? left : right;
#endif

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset2)) ? left : right;
#else
		next = (v > (th + offset2)) ? left : right;
#endif

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset2)) ? left : right;
#else
		next = (v > (th + offset2)) ? left : right;
#endif

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset2)) ? left : right;
#else
		next = (v > (th + offset2)) ? left : right;
#endif

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset2)) ? left : right;
#else
		next = (v > (th + offset2)) ? left : right;
#endif

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset2)) ? left : right;
#else
		next = (v > (th + offset2)) ? left : right;
#endif

		vR[i] = table[next] + offset2;
#endif

#if defined(_RF_TABLE_4TREE_CONST)
#ifdef INDIRECT
		v[0] = input[i];
#else
		v = input[i];
#endif
		next = 0;
#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset3)) ? left : right;
#else
		next = (v > (th + offset3)) ? left : right;
#endif

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset3)) ? left : right;
#else
		next = (v > (th + offset3)) ? left : right;
#endif

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset3)) ? left : right;
#else
		next = (v > (th + offset3)) ? left : right;
#endif

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset3)) ? left : right;
#else
		next = (v > (th + offset3)) ? left : right;
#endif

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset3)) ? left : right;
#else
		next = (v > (th + offset3)) ? left : right;
#endif

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset3)) ? left : right;
#else
		next = (v > (th + offset3)) ? left : right;
#endif

		vR[i] = table[next] + offset3;
#endif
	}
}

void initData(float *ip, int size);
void registerTime(float value);

int main(int argc, char **argv)
{
	float elapsed_time;
	// set up device
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("[CUDA - Random Forest]: EXP %s\n", EXP_NAME);
#ifdef INDIRECT
	printf("[CUDA - Random Forest]: INDIRECT\n");
#endif
	printf("[CUDA - Random Forest]: Using Device %d: %s\n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));

	// set up data size of vectors
	int nElem = N_ELEM;
	int tamTable = TAM_TABLE;
	int tamTh = TAM_TH;
	printf("[CUDA - Random Forest]: Vector Size %d\n", nElem);

	// malloc host memory
	size_t nBytes = nElem * sizeof(float);
	size_t tableBytes = tamTable * sizeof(int);
	size_t thBytes = tamTh * sizeof(float);

	float *h_vR;
	h_vR = (float *)malloc(nBytes);
	float *h_input;
	h_input = (float *)malloc(nBytes);
	int *h_table;
	h_table = (int *)malloc(tableBytes);
	float *h_th;
	h_th = (float *)malloc(thBytes);
	{
		h_table[0] = (2 << 2) | (1 << 10);
		h_table[1] = (4 << 2) | (3 << 10);
		h_table[2] = (6 << 2) | (5 << 10);
		h_table[3] = (8 << 2) | (7 << 10);
		h_table[4] = (10 << 2) | (9 << 10);
		h_table[5] = (12 << 2) | (11 << 10);
		h_table[6] = (14 << 2) | (13 << 10);
		h_table[7] = (16 << 2) | (15 << 10);
		h_table[8] = (18 << 2) | (17 << 10);
		h_table[9] = (20 << 2) | (19 << 10);
		h_table[10] = (22 << 2) | (21 << 10);
		h_table[11] = (24 << 2) | (23 << 10);
		h_table[12] = (26 << 2) | (25 << 10);
		h_table[13] = (28 << 2) | (27 << 10);
		h_table[14] = (30 << 2) | (29 << 10);
		h_table[15] = (32 << 2) | (31 << 10);
		h_table[16] = (34 << 2) | (33 << 10);
		h_table[17] = (36 << 2) | (35 << 10);
		h_table[18] = (38 << 2) | (37 << 10);
		h_table[19] = (40 << 2) | (39 << 10);
		h_table[20] = (42 << 2) | (41 << 10);
		h_table[21] = (44 << 2) | (43 << 10);
		h_table[22] = (46 << 2) | (45 << 10);
		h_table[23] = (48 << 2) | (47 << 10);
		h_table[24] = (50 << 2) | (49 << 10);
		h_table[25] = (52 << 2) | (51 << 10);
		h_table[26] = (54 << 2) | (53 << 10);
		h_table[27] = (56 << 2) | (55 << 10);
		h_table[28] = (58 << 2) | (57 << 10);
		h_table[29] = (60 << 2) | (59 << 10);
		h_table[30] = (62 << 2) | (61 << 10);
		h_table[31] = (64 << 2) | (63 << 10);
		h_table[32] = (66 << 2) | (65 << 10);
		h_table[33] = (68 << 2) | (67 << 10);
		h_table[34] = (70 << 2) | (69 << 10);
		h_table[35] = (72 << 2) | (71 << 10);
		h_table[36] = (74 << 2) | (73 << 10);
		h_table[37] = (76 << 2) | (75 << 10);
		h_table[38] = (78 << 2) | (77 << 10);
		h_table[39] = (80 << 2) | (79 << 10);
		h_table[40] = (82 << 2) | (81 << 10);
		h_table[41] = (84 << 2) | (83 << 10);
		h_table[42] = (86 << 2) | (85 << 10);
		h_table[43] = (88 << 2) | (87 << 10);
		h_table[44] = (90 << 2) | (89 << 10);
		h_table[45] = (92 << 2) | (91 << 10);
		h_table[46] = (94 << 2) | (93 << 10);
		h_table[47] = (96 << 2) | (95 << 10);
		h_table[48] = (98 << 2) | (97 << 10);
		h_table[49] = (100 << 2) | (99 << 10);
		h_table[50] = (102 << 2) | (101 << 10);
		h_table[51] = (104 << 2) | (103 << 10);
		h_table[52] = (106 << 2) | (105 << 10);
		h_table[53] = (108 << 2) | (107 << 10);
		h_table[54] = (110 << 2) | (109 << 10);
		h_table[55] = (112 << 2) | (111 << 10);
		h_table[56] = (114 << 2) | (113 << 10);
		h_table[57] = (116 << 2) | (115 << 10);
		h_table[58] = (118 << 2) | (117 << 10);
		h_table[59] = (120 << 2) | (119 << 10);
		h_table[60] = (122 << 2) | (121 << 10);
		h_table[61] = (124 << 2) | (123 << 10);
		h_table[62] = (126 << 2) | (125 << 10);
		h_table[63] = (128 << 2) | (127 << 10);
		h_table[64] = (130 << 2) | (129 << 10);
		h_table[65] = (132 << 2) | (131 << 10);
		h_table[66] = (134 << 2) | (133 << 10);
		h_table[67] = (136 << 2) | (135 << 10);
		h_table[68] = (138 << 2) | (137 << 10);
		h_table[69] = (140 << 2) | (139 << 10);
		h_table[70] = (142 << 2) | (141 << 10);
		h_table[71] = (144 << 2) | (143 << 10);
		h_table[72] = (146 << 2) | (145 << 10);
		h_table[73] = (148 << 2) | (147 << 10);
		h_table[74] = (150 << 2) | (149 << 10);
		h_table[75] = (152 << 2) | (151 << 10);
		h_table[76] = (154 << 2) | (153 << 10);
		h_table[77] = (156 << 2) | (155 << 10);
		h_table[78] = (158 << 2) | (157 << 10);
		h_table[79] = (160 << 2) | (159 << 10);
		h_table[80] = (162 << 2) | (161 << 10);
		h_table[81] = (164 << 2) | (163 << 10);
		h_table[82] = (166 << 2) | (165 << 10);
		h_table[83] = (168 << 2) | (167 << 10);
		h_table[84] = (170 << 2) | (169 << 10);
		h_table[85] = (172 << 2) | (171 << 10);
		h_table[86] = (174 << 2) | (173 << 10);
		h_table[87] = (176 << 2) | (175 << 10);
		h_table[88] = (178 << 2) | (177 << 10);
		h_table[89] = (180 << 2) | (179 << 10);
		h_table[90] = (182 << 2) | (181 << 10);
		h_table[91] = (184 << 2) | (183 << 10);
		h_table[92] = (186 << 2) | (185 << 10);
		h_table[93] = (188 << 2) | (187 << 10);
		h_table[94] = (190 << 2) | (189 << 10);
		h_table[95] = (192 << 2) | (191 << 10);
		h_table[96] = (194 << 2) | (193 << 10);
		h_table[97] = (196 << 2) | (195 << 10);
		h_table[98] = (198 << 2) | (197 << 10);
		h_table[99] = (200 << 2) | (199 << 10);
		h_table[100] = (202 << 2) | (201 << 10);
		h_table[101] = (204 << 2) | (203 << 10);
		h_table[102] = (206 << 2) | (205 << 10);
		h_table[103] = (208 << 2) | (207 << 10);
		h_table[104] = (210 << 2) | (209 << 10);
		h_table[105] = (212 << 2) | (211 << 10);
		h_table[106] = (214 << 2) | (213 << 10);
		h_table[107] = (216 << 2) | (215 << 10);
		h_table[108] = (218 << 2) | (217 << 10);
		h_table[109] = (220 << 2) | (219 << 10);
		h_table[110] = (222 << 2) | (221 << 10);
		h_table[111] = (224 << 2) | (223 << 10);
		h_table[112] = (226 << 2) | (225 << 10);
		h_table[113] = (228 << 2) | (227 << 10);
		h_table[114] = (230 << 2) | (229 << 10);
		h_table[115] = (232 << 2) | (231 << 10);
		h_table[116] = (234 << 2) | (233 << 10);
		h_table[117] = (236 << 2) | (235 << 10);
		h_table[118] = (238 << 2) | (237 << 10);
		h_table[119] = (240 << 2) | (239 << 10);
		h_table[120] = (242 << 2) | (241 << 10);
		h_table[121] = (244 << 2) | (243 << 10);
		h_table[122] = (246 << 2) | (245 << 10);
		h_table[123] = (248 << 2) | (247 << 10);
		h_table[124] = (250 << 2) | (249 << 10);
		h_table[125] = (252 << 2) | (251 << 10);
		h_table[126] = (254 << 2) | (253 << 10);
		h_table[127] = 0;
		h_table[128] = 1;
		h_table[129] = 2;
		h_table[130] = 3;
		h_table[131] = 4;
		h_table[132] = 5;
		h_table[133] = 6;
		h_table[134] = 7;
		h_table[135] = 8;
		h_table[136] = 9;
		h_table[137] = 10;
		h_table[138] = 11;
		h_table[139] = 12;
		h_table[140] = 13;
		h_table[141] = 14;
		h_table[142] = 15;
		h_table[143] = 16;
		h_table[144] = 17;
		h_table[145] = 18;
		h_table[146] = 19;
		h_table[147] = 20;
		h_table[148] = 21;
		h_table[149] = 22;
		h_table[150] = 23;
		h_table[151] = 24;
		h_table[152] = 25;
		h_table[153] = 26;
		h_table[154] = 27;
		h_table[155] = 28;
		h_table[156] = 29;
		h_table[157] = 30;
		h_table[158] = 31;
		h_table[159] = 32;
		h_table[160] = 33;
		h_table[161] = 34;
		h_table[162] = 35;
		h_table[163] = 36;
		h_table[164] = 37;
		h_table[165] = 38;
		h_table[166] = 39;
		h_table[167] = 40;
		h_table[168] = 41;
		h_table[169] = 42;
		h_table[170] = 43;
		h_table[171] = 44;
		h_table[172] = 45;
		h_table[173] = 46;
		h_table[174] = 47;
		h_table[175] = 48;
		h_table[176] = 49;
		h_table[177] = 50;
		h_table[178] = 51;
		h_table[179] = 52;
		h_table[180] = 53;
		h_table[181] = 54;
		h_table[182] = 55;
		h_table[183] = 56;
		h_table[184] = 57;
		h_table[185] = 58;
		h_table[186] = 59;
		h_table[187] = 60;
		h_table[188] = 61;
		h_table[189] = 62;
		h_table[190] = 63;
		h_table[191] = 64;
		h_table[192] = 65;
		h_table[193] = 66;
		h_table[194] = 67;
		h_table[195] = 68;
		h_table[196] = 69;
		h_table[197] = 70;
		h_table[198] = 71;
		h_table[199] = 72;
		h_table[200] = 73;
		h_table[201] = 74;
		h_table[202] = 75;
		h_table[203] = 76;
		h_table[204] = 77;
		h_table[205] = 78;
		h_table[206] = 79;
		h_table[207] = 80;
		h_table[208] = 81;
		h_table[209] = 82;
		h_table[210] = 83;
		h_table[211] = 84;
		h_table[212] = 85;
		h_table[213] = 86;
		h_table[214] = 87;
		h_table[215] = 88;
		h_table[216] = 89;
		h_table[217] = 90;
		h_table[218] = 91;
		h_table[219] = 92;
		h_table[220] = 93;
		h_table[221] = 94;
		h_table[222] = 95;
		h_table[223] = 96;
		h_table[224] = 97;
		h_table[225] = 98;
		h_table[226] = 99;
		h_table[227] = 100;
		h_table[228] = 101;
		h_table[229] = 102;
		h_table[230] = 103;
		h_table[231] = 104;
		h_table[232] = 105;
		h_table[233] = 106;
		h_table[234] = 107;
		h_table[235] = 108;
		h_table[236] = 109;
		h_table[237] = 110;
		h_table[238] = 111;
		h_table[239] = 112;
		h_table[240] = 113;
		h_table[241] = 114;
		h_table[242] = 115;
		h_table[243] = 116;
		h_table[244] = 117;
		h_table[245] = 118;
		h_table[246] = 119;
		h_table[247] = 120;
		h_table[248] = 121;
		h_table[249] = 122;
		h_table[250] = 123;
		h_table[251] = 124;
		h_table[252] = 125;
		h_table[253] = 126;
		h_table[254] = 127;

		h_th[0] = 63;
		h_th[1] = 31;
		h_th[2] = 95;
		h_th[3] = 15;
		h_th[4] = 47;
		h_th[5] = 79;
		h_th[6] = 111;
		h_th[7] = 7;
		h_th[8] = 23;
		h_th[9] = 39;
		h_th[10] = 55;
		h_th[11] = 71;
		h_th[12] = 87;
		h_th[13] = 103;
		h_th[14] = 119;
		h_th[15] = 3;
		h_th[16] = 11;
		h_th[17] = 19;
		h_th[18] = 27;
		h_th[19] = 35;
		h_th[20] = 43;
		h_th[21] = 51;
		h_th[22] = 59;
		h_th[23] = 67;
		h_th[24] = 75;
		h_th[25] = 83;
		h_th[26] = 91;
		h_th[27] = 99;
		h_th[28] = 107;
		h_th[29] = 115;
		h_th[30] = 123;
		h_th[31] = 1;
		h_th[32] = 5;
		h_th[33] = 9;
		h_th[34] = 13;
		h_th[35] = 17;
		h_th[36] = 21;
		h_th[37] = 25;
		h_th[38] = 29;
		h_th[39] = 33;
		h_th[40] = 37;
		h_th[41] = 41;
		h_th[42] = 45;
		h_th[43] = 49;
		h_th[44] = 53;
		h_th[45] = 57;
		h_th[46] = 61;
		h_th[47] = 65;
		h_th[48] = 69;
		h_th[49] = 73;
		h_th[50] = 77;
		h_th[51] = 81;
		h_th[52] = 85;
		h_th[53] = 89;
		h_th[54] = 93;
		h_th[55] = 97;
		h_th[56] = 101;
		h_th[57] = 105;
		h_th[58] = 109;
		h_th[59] = 113;
		h_th[60] = 117;
		h_th[61] = 121;
		h_th[62] = 125;
		h_th[63] = 1;
		h_th[64] = 3;
		h_th[65] = 5;
		h_th[66] = 7;
		h_th[67] = 9;
		h_th[68] = 11;
		h_th[69] = 13;
		h_th[70] = 15;
		h_th[71] = 17;
		h_th[72] = 19;
		h_th[73] = 21;
		h_th[74] = 23;
		h_th[75] = 25;
		h_th[76] = 27;
		h_th[77] = 29;
		h_th[78] = 31;
		h_th[79] = 33;
		h_th[80] = 35;
		h_th[81] = 37;
		h_th[82] = 39;
		h_th[83] = 41;
		h_th[84] = 43;
		h_th[85] = 45;
		h_th[86] = 47;
		h_th[87] = 49;
		h_th[88] = 51;
		h_th[89] = 53;
		h_th[90] = 55;
		h_th[91] = 57;
		h_th[92] = 59;
		h_th[93] = 61;
		h_th[94] = 63;
		h_th[95] = 65;
		h_th[96] = 67;
		h_th[97] = 69;
		h_th[98] = 71;
		h_th[99] = 73;
		h_th[100] = 75;
		h_th[101] = 77;
		h_th[102] = 79;
		h_th[103] = 81;
		h_th[104] = 83;
		h_th[105] = 85;
		h_th[106] = 87;
		h_th[107] = 89;
		h_th[108] = 91;
		h_th[109] = 93;
		h_th[110] = 95;
		h_th[111] = 97;
		h_th[112] = 99;
		h_th[113] = 101;
		h_th[114] = 103;
		h_th[115] = 105;
		h_th[116] = 107;
		h_th[117] = 109;
		h_th[118] = 111;
		h_th[119] = 113;
		h_th[120] = 115;
		h_th[121] = 117;
		h_th[122] = 119;
		h_th[123] = 121;
		h_th[124] = 123;
		h_th[125] = 125;
		h_th[126] = 127;
	}
	printf("[CUDA - Random Forest]: Start Reading Dataset\n");
	initData(h_input, nElem);
	printf("[CUDA - Random Forest]: Finished Reading Dataset\n");
	memset(h_vR, 0, nBytes);

	// malloc device global memory
	float *d_vR;
	CHECK(hipMalloc((float **)&d_vR, nBytes));
	float *d_input;
	CHECK(hipMalloc((float **)&d_input, nBytes));

	// transfer data from host to device
	CHECK(hipMemcpy(d_input, h_input, nBytes, hipMemcpyHostToDevice));

	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(table), h_table, TAM_TABLE * sizeof(int)));
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(TH), h_th, TAM_TH * sizeof(float)));

	// invoke kernel at host side
	int iLen = 512;
	dim3 block(iLen);
	dim3 grid((nElem + block.x - 1) / block.x);
	hipEvent_t start, stop;
	CHECK(hipEventCreate(&start));
	CHECK(hipEventCreate(&stop));

	// record start event
	CHECK(hipEventRecord(start, 0));

	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, d_vR);

	CHECK(hipEventRecord(stop, 0));
	CHECK(hipEventSynchronize(stop));
	hipDeviceSynchronize();

	// calculate elapsed time
	CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
	printf("[CUDA - Random Forest]: %s - execution time = %.6fms\n", EXP_NAME, elapsed_time);
	registerTime(elapsed_time);

	// copy kernel result back to host side
	CHECK(hipMemcpy(h_vR, d_vR, nBytes, hipMemcpyDeviceToHost));
	CHECK(hipGetLastError());

	// Verify the answers
	char flag = 1;
	for (int i = 0; i < nElem; i++)
	{
		if (i < 32)
		{
			printf("%.3f ", h_vR[i]);
		}
		if (h_input[i] != h_vR[i])
		{
			flag = 0;
			// break;
		}
	}
	if (flag == 1)
	{
		printf("Return correct");
	}
	else
	{
		printf("Error!!");
	}
	printf("\n ");

	// free device memory
	hipFree(d_input);
	hipFree(d_vR);

	// free host memory
	free(h_input);
	free(h_vR);
	free(h_th);
	free(h_table);
	return (0);
}

void initData(float *ip, int size)
{
	for (int i = 0; i < size; i++)
	{
		if (WARP_SIZE < _MOD)
		{
			int aux = _MOD / WARP_SIZE;
			ip[i] = (float)((i * aux) % _MOD);
		}
		else
		{
			ip[i] = (float)(i % _MOD);
		}
		// if (i < 32)
		//	printf("%.2f ", ip[i]);
	}
	return;
}

void registerTime(float value)
{
	printf("[CUDA - Random Forest]: Time spent: %f.\n", value);
}
