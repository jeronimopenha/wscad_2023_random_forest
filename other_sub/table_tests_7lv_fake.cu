#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>

#define TAM_TABLE 15
#define TAM_TH 7

//#define N_ELEM 50000000
#define N_ELEM 100000000

// #define _RF_TABLE_1TREE_7LV_FAKE
//  #define _RF_TABLE_2TREE_7LV_FAKE
//  #define _RF_TABLE_3TREE_7LV_FAKE
//  #define _RF_TABLE_4TREE_7LV_FAKE

#define WARP_SIZE 32

#define CHECK(call)                                                \
	{                                                              \
		const hipError_t error = call;                            \
		if (error != hipSuccess)                                  \
		{                                                          \
			fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
			fprintf(stderr, "code: %d, reason: %s\n", error,       \
					hipGetErrorString(error));                    \
		}                                                          \
	}

#define _MOD 8

#define _1TREE
#define EXP_NAME "_RF_TABLE_1TREE_7LV_FAKE"
__global__ void RF_kernel(float *input, const int n, const int *table, const int offset0, float *vR)

{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int next, left, right;
	float th;
	int tb[TAM_TABLE];

	int v;

	if (i < n)
	{

		for (int j = 0; j < TAM_TABLE; j++)
		{
			tb[j] = table[j];
		}

		v = input[i];
		left = (tb[0] >> 2) & 0xff;
		right = (tb[1] >> 10) & 0xff;
		th = tb[2];
		next = (v > (th + offset0)) ? left : right;

		// if (i == 2)
		//	printf("idx=%d, left=%d , right=%d , th=%f , next=%d \n", idx, left, right, th, next);

		left = (tb[3] >> 2) & 0xff;
		right = (tb[4] >> 10) & 0xff;
		th = tb[5];
		next += (v > (th + offset0)) ? left : right;

		left = (tb[6] >> 2) & 0xff;
		right = (tb[7] >> 10) & 0xff;
		th = tb[8];
		next += (v > (th + offset0)) ? left : right;

		vR[i] = next + offset0;
    }
}

void initData(float *ip, int size);
void registerTime(float value);

int main(int argc, char **argv)
{
	float elapsed_time;
	// set up device
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("[CUDA - Random Forest]: EXP %s\n", EXP_NAME);
	printf("[CUDA - Random Forest]: Using Device %d: %s\n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));

	// set up data size of vectors
	int nElem = N_ELEM;
	int tamTable = TAM_TABLE;
	int tamTh = TAM_TH;
	printf("[CUDA - Random Forest]: Vector Size %d\n", nElem);

	// malloc host memory
	size_t nBytes = nElem * sizeof(float);
	size_t tableBytes = tamTable * sizeof(int);
	size_t thBytes = tamTh * sizeof(float);

	float *h_vR;
	h_vR = (float *)malloc(nBytes);
	float *h_input;
	h_input = (float *)malloc(nBytes);
	int *h_table;
	h_table = (int *)malloc(tableBytes);
	float *h_th;
	h_th = (float *)malloc(thBytes);
	{
		h_table[0] = (1 << 2) | (2 << 10);
		h_table[1] = (3 << 2) | (4 << 10);
		h_table[2] = (5 << 2) | (6 << 10);
		h_table[3] = (7 << 2) | (8 << 10);
		h_table[4] = (9 << 2) | (10 << 10);
		h_table[5] = (11 << 2) | (12 << 10);
		h_table[6] = (13 << 2) | (14 << 10);
		h_table[7] = 7;
		h_table[8] = 6;
		h_table[9] = 5;
		h_table[10] = 4;
		h_table[11] = 3;
		h_table[12] = 2;
		h_table[13] = 1;
		h_table[14] = 0;
	}
	printf("[CUDA - Random Forest]: Start Reading Dataset\n");
	initData(h_input, nElem);
	printf("[CUDA - Random Forest]: Finished Reading Dataset\n");
	memset(h_vR, 0, nBytes);

	// malloc device global memory
	float *d_vR;
	CHECK(hipMalloc((float **)&d_vR, nBytes));
	float *d_input;
	CHECK(hipMalloc((float **)&d_input, nBytes));
	int *d_table;
	CHECK(hipMalloc((int **)&d_table, tableBytes));

	// transfer data from host to device
	CHECK(hipMemcpy(d_input, h_input, nBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_table, h_table, tableBytes, hipMemcpyHostToDevice));

	// invoke kernel at host side
	int iLen = 512;
	dim3 block(iLen);
	dim3 grid((nElem + block.x - 1) / block.x);
	hipEvent_t start, stop;
	CHECK(hipEventCreate(&start));
	CHECK(hipEventCreate(&stop));

	// record start event
	CHECK(hipEventRecord(start, 0));

#if defined(_2TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, d_vR);
#elif defined(_3TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, d_vR);
#elif defined(_4TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, d_vR);
#else
	RF_kernel<<<grid, block>>>(d_input, nElem, d_table, 0, d_vR);
#endif

	CHECK(hipEventRecord(stop, 0));
	CHECK(hipEventSynchronize(stop));
	hipDeviceSynchronize();

	// calculate elapsed time
	CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
	printf("[CUDA - Random Forest]: %s - execution time = %.6fms\n", EXP_NAME, elapsed_time);
	registerTime(elapsed_time);

	// copy kernel result back to host side
	CHECK(hipMemcpy(h_vR, d_vR, nBytes, hipMemcpyDeviceToHost));
	CHECK(hipGetLastError());

	// Verify the answers
	char flag = 1;
	for (int i = 0; i < nElem; i++)
	{
		if (i < 4 * WARP_SIZE)
		{
			printf("%.3f ", h_vR[i]);
		}
		if (h_input[i] != h_vR[i])
		{
			flag = 0;
			// break;
		}
	}
	if (flag == 1)
	{
		// printf("Return correct");
	}
	else
	{
		// printf("Error!!");
	}
	printf("\n ");

	// free device memory
	hipFree(d_input);
	hipFree(d_vR);

	// free host memory
	free(h_input);
	free(h_vR);
	free(h_th);
	free(h_table);
	return (0);
}

void initData(float *ip, int size)
{
	for (int i = 0; i < size; i++)
	{
		ip[i] = (float)0;
		// if (WARP_SIZE < _MOD)
		//{
		//	int aux = _MOD / WARP_SIZE;
		//	ip[i] = (float)((i * aux) % _MOD);
		// }
		// else
		//{
		//	ip[i] = (float)(i % _MOD);
		// }
		//  if (i < 32)
		//	printf("%.2f ", ip[i]);
	}
	return;
}

void registerTime(float value)
{
	printf("[CUDA - Random Forest]: Time spent: %f.\n", value);
}
