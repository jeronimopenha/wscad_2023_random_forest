#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>

#define OFFSET_SIZE 10
#define N_ELEM 50000000
// #define N_ELEM 100000000
#define WARP_SIZE 32

#define CHECK(call)                                                \
    {                                                              \
        const hipError_t error = call;                            \
        if (error != hipSuccess)                                  \
        {                                                          \
            fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
            fprintf(stderr, "code: %d, reason: %s\n", error,       \
                    hipGetErrorString(error));                    \
        }                                                          \
    }

#define _MOD 32
#define _1TREE
#define EXP_NAME "_RF_IF_10TREE_5LV_2_2"

__global__ void RF_kernel(float *input, const int n, const int *offset, float *vR)
{
    __shared__ int temp[1024];

    int i = blockIdx.x * blockDim.x / 2 + threadIdx.x % 512;
    // rodar os de prof 5 e 6 com 4 árvores

    if (i < n)
    {
        if (threadIdx.x < 512)
        {
            if (input[i] > (15 + offset[0]))
            {
                if (input[i] > (23 + offset[0]))
                {
                    if (input[i] > (27 + offset[0]))
                    {
                        if (input[i] > (29 + offset[0]))
                        {
                            if (input[i] == (31 + offset[0]))
                            {
                                temp[threadIdx.x] = 31;
                            }
                            else
                            {
                                temp[threadIdx.x] = 30;
                            }
                        }
                        else
                        {
                            if (input[i] == (29 + offset[0]))
                            {
                                temp[threadIdx.x] = 29;
                            }
                            else
                            {
                                temp[threadIdx.x] = 28;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (25 + offset[0]))
                        {
                            if (input[i] == (27 + offset[0]))
                            {
                                temp[threadIdx.x] = 27;
                            }
                            else
                            {
                                temp[threadIdx.x] = 26;
                            }
                        }
                        else
                        {
                            if (input[i] == (25 + offset[0]))
                            {
                                temp[threadIdx.x] = 25;
                            }
                            else
                            {
                                temp[threadIdx.x] = 24;
                            }
                        }
                    }
                }
                else
                {
                    if (input[i] > (19 + offset[0]))
                    {
                        if (input[i] > (21 + offset[0]))
                        {
                            if (input[i] == (23 + offset[0]))
                            {
                                temp[threadIdx.x] = 23;
                            }
                            else
                            {
                                temp[threadIdx.x] = 22;
                            }
                        }
                        else
                        {
                            if (input[i] == (21 + offset[0]))
                            {
                                temp[threadIdx.x] = 21;
                            }
                            else
                            {
                                temp[threadIdx.x] = 20;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (17 + offset[0]))
                        {
                            if (input[i] == (19 + offset[0]))
                            {
                                temp[threadIdx.x] = 19;
                            }
                            else
                            {
                                temp[threadIdx.x] = 18;
                            }
                        }
                        else
                        {
                            if (input[i] == (17 + offset[0]))
                            {
                                temp[threadIdx.x] = 17;
                            }
                            else
                            {
                                temp[threadIdx.x] = 16;
                            }
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (7 + offset[0]))
                {
                    if (input[i] > (11 + offset[0]))
                    {
                        if (input[i] > (13 + offset[0]))
                        {
                            if (input[i] == (15 + offset[0]))
                            {
                                temp[threadIdx.x] = 15;
                            }
                            else
                            {
                                temp[threadIdx.x] = 14;
                            }
                        }
                        else
                        {
                            if (input[i] == (13 + offset[0]))
                            {
                                temp[threadIdx.x] = 13;
                            }
                            else
                            {
                                temp[threadIdx.x] = 12;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (9 + offset[0]))
                        {
                            if (input[i] == (11 + offset[0]))
                            {
                                temp[threadIdx.x] = 11;
                            }
                            else
                            {
                                temp[threadIdx.x] = 10;
                            }
                        }
                        else
                        {
                            if (input[i] == (9 + offset[0]))
                            {
                                temp[threadIdx.x] = 9;
                            }
                            else
                            {
                                temp[threadIdx.x] = 8;
                            }
                        }
                    }
                }
                else
                {
                    if (input[i] > (3 + offset[0]))
                    {
                        if (input[i] > (5 + offset[0]))
                        {
                            if (input[i] == (7 + offset[0]))
                            {
                                temp[threadIdx.x] = 7;
                            }
                            else
                            {
                                temp[threadIdx.x] = 6;
                            }
                        }
                        else
                        {
                            if (input[i] == (5 + offset[0]))
                            {
                                temp[threadIdx.x] = 5;
                            }
                            else
                            {
                                temp[threadIdx.x] = 4;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (1 + offset[0]))
                        {
                            if (input[i] == (3 + offset[0]))
                            {
                                temp[threadIdx.x] = 3;
                            }
                            else
                            {
                                temp[threadIdx.x] = 2;
                            }
                        }
                        else
                        {
                            if (input[i] == (1 + offset[0]))
                            {
                                temp[threadIdx.x] = 1;
                            }
                            else
                            {
                                temp[threadIdx.x] = 0;
                            }
                        }
                    }
                }
            }

            if (input[i] > (15 + offset[1]))
            {
                if (input[i] > (23 + offset[1]))
                {
                    if (input[i] > (27 + offset[1]))
                    {
                        if (input[i] > (29 + offset[1]))
                        {
                            if (input[i] == (31 + offset[1]))
                            {
                                temp[threadIdx.x] = 31;
                            }
                            else
                            {
                                temp[threadIdx.x] = 30;
                            }
                        }
                        else
                        {
                            if (input[i] == (29 + offset[1]))
                            {
                                temp[threadIdx.x] = 29;
                            }
                            else
                            {
                                temp[threadIdx.x] = 28;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (25 + offset[1]))
                        {
                            if (input[i] == (27 + offset[1]))
                            {
                                temp[threadIdx.x] = 27;
                            }
                            else
                            {
                                temp[threadIdx.x] = 26;
                            }
                        }
                        else
                        {
                            if (input[i] == (25 + offset[1]))
                            {
                                temp[threadIdx.x] = 25;
                            }
                            else
                            {
                                temp[threadIdx.x] = 24;
                            }
                        }
                    }
                }
                else
                {
                    if (input[i] > (19 + offset[1]))
                    {
                        if (input[i] > (21 + offset[1]))
                        {
                            if (input[i] == (23 + offset[1]))
                            {
                                temp[threadIdx.x] = 23;
                            }
                            else
                            {
                                temp[threadIdx.x] = 22;
                            }
                        }
                        else
                        {
                            if (input[i] == (21 + offset[1]))
                            {
                                temp[threadIdx.x] = 21;
                            }
                            else
                            {
                                temp[threadIdx.x] = 20;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (17 + offset[1]))
                        {
                            if (input[i] == (19 + offset[1]))
                            {
                                temp[threadIdx.x] = 19;
                            }
                            else
                            {
                                temp[threadIdx.x] = 18;
                            }
                        }
                        else
                        {
                            if (input[i] == (17 + offset[1]))
                            {
                                temp[threadIdx.x] = 17;
                            }
                            else
                            {
                                temp[threadIdx.x] = 16;
                            }
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (7 + offset[1]))
                {
                    if (input[i] > (11 + offset[1]))
                    {
                        if (input[i] > (13 + offset[1]))
                        {
                            if (input[i] == (15 + offset[1]))
                            {
                                temp[threadIdx.x] = 15;
                            }
                            else
                            {
                                temp[threadIdx.x] = 14;
                            }
                        }
                        else
                        {
                            if (input[i] == (13 + offset[1]))
                            {
                                temp[threadIdx.x] = 13;
                            }
                            else
                            {
                                temp[threadIdx.x] = 12;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (9 + offset[1]))
                        {
                            if (input[i] == (11 + offset[1]))
                            {
                                temp[threadIdx.x] = 11;
                            }
                            else
                            {
                                temp[threadIdx.x] = 10;
                            }
                        }
                        else
                        {
                            if (input[i] == (9 + offset[1]))
                            {
                                temp[threadIdx.x] = 9;
                            }
                            else
                            {
                                temp[threadIdx.x] = 8;
                            }
                        }
                    }
                }
                else
                {
                    if (input[i] > (3 + offset[1]))
                    {
                        if (input[i] > (5 + offset[1]))
                        {
                            if (input[i] == (7 + offset[1]))
                            {
                                temp[threadIdx.x] = 7;
                            }
                            else
                            {
                                temp[threadIdx.x] = 6;
                            }
                        }
                        else
                        {
                            if (input[i] == (5 + offset[1]))
                            {
                                temp[threadIdx.x] = 5;
                            }
                            else
                            {
                                temp[threadIdx.x] = 4;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (1 + offset[1]))
                        {
                            if (input[i] == (3 + offset[1]))
                            {
                                temp[threadIdx.x] = 3;
                            }
                            else
                            {
                                temp[threadIdx.x] = 2;
                            }
                        }
                        else
                        {
                            if (input[i] == (1 + offset[1]))
                            {
                                temp[threadIdx.x] = 1;
                            }
                            else
                            {
                                temp[threadIdx.x] = 0;
                            }
                        }
                    }
                }
            }

            if (input[i] > (15 + offset[2]))
            {
                if (input[i] > (23 + offset[2]))
                {
                    if (input[i] > (27 + offset[2]))
                    {
                        if (input[i] > (29 + offset[2]))
                        {
                            if (input[i] == (31 + offset[2]))
                            {
                                temp[threadIdx.x] = 31;
                            }
                            else
                            {
                                temp[threadIdx.x] = 30;
                            }
                        }
                        else
                        {
                            if (input[i] == (29 + offset[2]))
                            {
                                temp[threadIdx.x] = 29;
                            }
                            else
                            {
                                temp[threadIdx.x] = 28;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (25 + offset[2]))
                        {
                            if (input[i] == (27 + offset[2]))
                            {
                                temp[threadIdx.x] = 27;
                            }
                            else
                            {
                                temp[threadIdx.x] = 26;
                            }
                        }
                        else
                        {
                            if (input[i] == (25 + offset[2]))
                            {
                                temp[threadIdx.x] = 25;
                            }
                            else
                            {
                                temp[threadIdx.x] = 24;
                            }
                        }
                    }
                }
                else
                {
                    if (input[i] > (19 + offset[2]))
                    {
                        if (input[i] > (21 + offset[2]))
                        {
                            if (input[i] == (23 + offset[2]))
                            {
                                temp[threadIdx.x] = 23;
                            }
                            else
                            {
                                temp[threadIdx.x] = 22;
                            }
                        }
                        else
                        {
                            if (input[i] == (21 + offset[2]))
                            {
                                temp[threadIdx.x] = 21;
                            }
                            else
                            {
                                temp[threadIdx.x] = 20;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (17 + offset[2]))
                        {
                            if (input[i] == (19 + offset[2]))
                            {
                                temp[threadIdx.x] = 19;
                            }
                            else
                            {
                                temp[threadIdx.x] = 18;
                            }
                        }
                        else
                        {
                            if (input[i] == (17 + offset[2]))
                            {
                                temp[threadIdx.x] = 17;
                            }
                            else
                            {
                                temp[threadIdx.x] = 16;
                            }
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (7 + offset[2]))
                {
                    if (input[i] > (11 + offset[2]))
                    {
                        if (input[i] > (13 + offset[2]))
                        {
                            if (input[i] == (15 + offset[2]))
                            {
                                temp[threadIdx.x] = 15;
                            }
                            else
                            {
                                temp[threadIdx.x] = 14;
                            }
                        }
                        else
                        {
                            if (input[i] == (13 + offset[2]))
                            {
                                temp[threadIdx.x] = 13;
                            }
                            else
                            {
                                temp[threadIdx.x] = 12;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (9 + offset[2]))
                        {
                            if (input[i] == (11 + offset[2]))
                            {
                                temp[threadIdx.x] = 11;
                            }
                            else
                            {
                                temp[threadIdx.x] = 10;
                            }
                        }
                        else
                        {
                            if (input[i] == (9 + offset[2]))
                            {
                                temp[threadIdx.x] = 9;
                            }
                            else
                            {
                                temp[threadIdx.x] = 8;
                            }
                        }
                    }
                }
                else
                {
                    if (input[i] > (3 + offset[2]))
                    {
                        if (input[i] > (5 + offset[2]))
                        {
                            if (input[i] == (7 + offset[2]))
                            {
                                temp[threadIdx.x] = 7;
                            }
                            else
                            {
                                temp[threadIdx.x] = 6;
                            }
                        }
                        else
                        {
                            if (input[i] == (5 + offset[2]))
                            {
                                temp[threadIdx.x] = 5;
                            }
                            else
                            {
                                temp[threadIdx.x] = 4;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (1 + offset[2]))
                        {
                            if (input[i] == (3 + offset[2]))
                            {
                                temp[threadIdx.x] = 3;
                            }
                            else
                            {
                                temp[threadIdx.x] = 2;
                            }
                        }
                        else
                        {
                            if (input[i] == (1 + offset[2]))
                            {
                                temp[threadIdx.x] = 1;
                            }
                            else
                            {
                                temp[threadIdx.x] = 0;
                            }
                        }
                    }
                }
            }

            if (input[i] > (15 + offset[3]))
            {
                if (input[i] > (23 + offset[3]))
                {
                    if (input[i] > (27 + offset[3]))
                    {
                        if (input[i] > (29 + offset[3]))
                        {
                            if (input[i] == (31 + offset[3]))
                            {
                                temp[threadIdx.x] = 31;
                            }
                            else
                            {
                                temp[threadIdx.x] = 30;
                            }
                        }
                        else
                        {
                            if (input[i] == (29 + offset[3]))
                            {
                                temp[threadIdx.x] = 29;
                            }
                            else
                            {
                                temp[threadIdx.x] = 28;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (25 + offset[3]))
                        {
                            if (input[i] == (27 + offset[3]))
                            {
                                temp[threadIdx.x] = 27;
                            }
                            else
                            {
                                temp[threadIdx.x] = 26;
                            }
                        }
                        else
                        {
                            if (input[i] == (25 + offset[3]))
                            {
                                temp[threadIdx.x] = 25;
                            }
                            else
                            {
                                temp[threadIdx.x] = 24;
                            }
                        }
                    }
                }
                else
                {
                    if (input[i] > (19 + offset[3]))
                    {
                        if (input[i] > (21 + offset[3]))
                        {
                            if (input[i] == (23 + offset[3]))
                            {
                                temp[threadIdx.x] = 23;
                            }
                            else
                            {
                                temp[threadIdx.x] = 22;
                            }
                        }
                        else
                        {
                            if (input[i] == (21 + offset[3]))
                            {
                                temp[threadIdx.x] = 21;
                            }
                            else
                            {
                                temp[threadIdx.x] = 20;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (17 + offset[3]))
                        {
                            if (input[i] == (19 + offset[3]))
                            {
                                temp[threadIdx.x] = 19;
                            }
                            else
                            {
                                temp[threadIdx.x] = 18;
                            }
                        }
                        else
                        {
                            if (input[i] == (17 + offset[3]))
                            {
                                temp[threadIdx.x] = 17;
                            }
                            else
                            {
                                temp[threadIdx.x] = 16;
                            }
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (7 + offset[3]))
                {
                    if (input[i] > (11 + offset[3]))
                    {
                        if (input[i] > (13 + offset[3]))
                        {
                            if (input[i] == (15 + offset[3]))
                            {
                                temp[threadIdx.x] = 15;
                            }
                            else
                            {
                                temp[threadIdx.x] = 14;
                            }
                        }
                        else
                        {
                            if (input[i] == (13 + offset[3]))
                            {
                                temp[threadIdx.x] = 13;
                            }
                            else
                            {
                                temp[threadIdx.x] = 12;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (9 + offset[3]))
                        {
                            if (input[i] == (11 + offset[3]))
                            {
                                temp[threadIdx.x] = 11;
                            }
                            else
                            {
                                temp[threadIdx.x] = 10;
                            }
                        }
                        else
                        {
                            if (input[i] == (9 + offset[3]))
                            {
                                temp[threadIdx.x] = 9;
                            }
                            else
                            {
                                temp[threadIdx.x] = 8;
                            }
                        }
                    }
                }
                else
                {
                    if (input[i] > (3 + offset[3]))
                    {
                        if (input[i] > (5 + offset[3]))
                        {
                            if (input[i] == (7 + offset[3]))
                            {
                                temp[threadIdx.x] = 7;
                            }
                            else
                            {
                                temp[threadIdx.x] = 6;
                            }
                        }
                        else
                        {
                            if (input[i] == (5 + offset[3]))
                            {
                                temp[threadIdx.x] = 5;
                            }
                            else
                            {
                                temp[threadIdx.x] = 4;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (1 + offset[3]))
                        {
                            if (input[i] == (3 + offset[3]))
                            {
                                temp[threadIdx.x] = 3;
                            }
                            else
                            {
                                temp[threadIdx.x] = 2;
                            }
                        }
                        else
                        {
                            if (input[i] == (1 + offset[3]))
                            {
                                temp[threadIdx.x] = 1;
                            }
                            else
                            {
                                temp[threadIdx.x] = 0;
                            }
                        }
                    }
                }
            }
            if (input[i] > (15 + offset[4]))
            {
                if (input[i] > (23 + offset[4]))
                {
                    if (input[i] > (27 + offset[4]))
                    {
                        if (input[i] > (29 + offset[4]))
                        {
                            if (input[i] == (31 + offset[4]))
                            {
                                temp[threadIdx.x] = 31;
                            }
                            else
                            {
                                temp[threadIdx.x] = 30;
                            }
                        }
                        else
                        {
                            if (input[i] == (29 + offset[4]))
                            {
                                temp[threadIdx.x] = 29;
                            }
                            else
                            {
                                temp[threadIdx.x] = 28;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (25 + offset[4]))
                        {
                            if (input[i] == (27 + offset[4]))
                            {
                                temp[threadIdx.x] = 27;
                            }
                            else
                            {
                                temp[threadIdx.x] = 26;
                            }
                        }
                        else
                        {
                            if (input[i] == (25 + offset[4]))
                            {
                                temp[threadIdx.x] = 25;
                            }
                            else
                            {
                                temp[threadIdx.x] = 24;
                            }
                        }
                    }
                }
                else
                {
                    if (input[i] > (19 + offset[4]))
                    {
                        if (input[i] > (21 + offset[4]))
                        {
                            if (input[i] == (23 + offset[4]))
                            {
                                temp[threadIdx.x] = 23;
                            }
                            else
                            {
                                temp[threadIdx.x] = 22;
                            }
                        }
                        else
                        {
                            if (input[i] == (21 + offset[4]))
                            {
                                temp[threadIdx.x] = 21;
                            }
                            else
                            {
                                temp[threadIdx.x] = 20;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (17 + offset[4]))
                        {
                            if (input[i] == (19 + offset[4]))
                            {
                                temp[threadIdx.x] = 19;
                            }
                            else
                            {
                                temp[threadIdx.x] = 18;
                            }
                        }
                        else
                        {
                            if (input[i] == (17 + offset[4]))
                            {
                                temp[threadIdx.x] = 17;
                            }
                            else
                            {
                                temp[threadIdx.x] = 16;
                            }
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (7 + offset[4]))
                {
                    if (input[i] > (11 + offset[4]))
                    {
                        if (input[i] > (13 + offset[4]))
                        {
                            if (input[i] == (15 + offset[4]))
                            {
                                temp[threadIdx.x] = 15;
                            }
                            else
                            {
                                temp[threadIdx.x] = 14;
                            }
                        }
                        else
                        {
                            if (input[i] == (13 + offset[4]))
                            {
                                temp[threadIdx.x] = 13;
                            }
                            else
                            {
                                temp[threadIdx.x] = 12;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (9 + offset[4]))
                        {
                            if (input[i] == (11 + offset[4]))
                            {
                                temp[threadIdx.x] = 11;
                            }
                            else
                            {
                                temp[threadIdx.x] = 10;
                            }
                        }
                        else
                        {
                            if (input[i] == (9 + offset[4]))
                            {
                                temp[threadIdx.x] = 9;
                            }
                            else
                            {
                                temp[threadIdx.x] = 8;
                            }
                        }
                    }
                }
                else
                {
                    if (input[i] > (3 + offset[4]))
                    {
                        if (input[i] > (5 + offset[4]))
                        {
                            if (input[i] == (7 + offset[4]))
                            {
                                temp[threadIdx.x] = 7;
                            }
                            else
                            {
                                temp[threadIdx.x] = 6;
                            }
                        }
                        else
                        {
                            if (input[i] == (5 + offset[4]))
                            {
                                temp[threadIdx.x] = 5;
                            }
                            else
                            {
                                temp[threadIdx.x] = 4;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (1 + offset[4]))
                        {
                            if (input[i] == (3 + offset[4]))
                            {
                                temp[threadIdx.x] = 3;
                            }
                            else
                            {
                                temp[threadIdx.x] = 2;
                            }
                        }
                        else
                        {
                            if (input[i] == (1 + offset[4]))
                            {
                                temp[threadIdx.x] = 1;
                            }
                            else
                            {
                                temp[threadIdx.x] = 0;
                            }
                        }
                    }
                }
            }
        }
        else
        {
            if (input[i] > (15 + offset[5]))
            {
                if (input[i] > (23 + offset[5]))
                {
                    if (input[i] > (27 + offset[5]))
                    {
                        if (input[i] > (29 + offset[5]))
                        {
                            if (input[i] == (31 + offset[5]))
                            {
                                temp[threadIdx.x] = 31;
                            }
                            else
                            {
                                temp[threadIdx.x] = 30;
                            }
                        }
                        else
                        {
                            if (input[i] == (29 + offset[5]))
                            {
                                temp[threadIdx.x] = 29;
                            }
                            else
                            {
                                temp[threadIdx.x] = 28;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (25 + offset[5]))
                        {
                            if (input[i] == (27 + offset[5]))
                            {
                                temp[threadIdx.x] = 27;
                            }
                            else
                            {
                                temp[threadIdx.x] = 26;
                            }
                        }
                        else
                        {
                            if (input[i] == (25 + offset[5]))
                            {
                                temp[threadIdx.x] = 25;
                            }
                            else
                            {
                                temp[threadIdx.x] = 24;
                            }
                        }
                    }
                }
                else
                {
                    if (input[i] > (19 + offset[5]))
                    {
                        if (input[i] > (21 + offset[5]))
                        {
                            if (input[i] == (23 + offset[5]))
                            {
                                temp[threadIdx.x] = 23;
                            }
                            else
                            {
                                temp[threadIdx.x] = 22;
                            }
                        }
                        else
                        {
                            if (input[i] == (21 + offset[5]))
                            {
                                temp[threadIdx.x] = 21;
                            }
                            else
                            {
                                temp[threadIdx.x] = 20;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (17 + offset[5]))
                        {
                            if (input[i] == (19 + offset[5]))
                            {
                                temp[threadIdx.x] = 19;
                            }
                            else
                            {
                                temp[threadIdx.x] = 18;
                            }
                        }
                        else
                        {
                            if (input[i] == (17 + offset[5]))
                            {
                                temp[threadIdx.x] = 17;
                            }
                            else
                            {
                                temp[threadIdx.x] = 16;
                            }
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (7 + offset[5]))
                {
                    if (input[i] > (11 + offset[5]))
                    {
                        if (input[i] > (13 + offset[5]))
                        {
                            if (input[i] == (15 + offset[5]))
                            {
                                temp[threadIdx.x] = 15;
                            }
                            else
                            {
                                temp[threadIdx.x] = 14;
                            }
                        }
                        else
                        {
                            if (input[i] == (13 + offset[5]))
                            {
                                temp[threadIdx.x] = 13;
                            }
                            else
                            {
                                temp[threadIdx.x] = 12;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (9 + offset[5]))
                        {
                            if (input[i] == (11 + offset[5]))
                            {
                                temp[threadIdx.x] = 11;
                            }
                            else
                            {
                                temp[threadIdx.x] = 10;
                            }
                        }
                        else
                        {
                            if (input[i] == (9 + offset[5]))
                            {
                                temp[threadIdx.x] = 9;
                            }
                            else
                            {
                                temp[threadIdx.x] = 8;
                            }
                        }
                    }
                }
                else
                {
                    if (input[i] > (3 + offset[5]))
                    {
                        if (input[i] > (5 + offset[5]))
                        {
                            if (input[i] == (7 + offset[5]))
                            {
                                temp[threadIdx.x] = 7;
                            }
                            else
                            {
                                temp[threadIdx.x] = 6;
                            }
                        }
                        else
                        {
                            if (input[i] == (5 + offset[5]))
                            {
                                temp[threadIdx.x] = 5;
                            }
                            else
                            {
                                temp[threadIdx.x] = 4;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (1 + offset[5]))
                        {
                            if (input[i] == (3 + offset[5]))
                            {
                                temp[threadIdx.x] = 3;
                            }
                            else
                            {
                                temp[threadIdx.x] = 2;
                            }
                        }
                        else
                        {
                            if (input[i] == (1 + offset[5]))
                            {
                                temp[threadIdx.x] = 1;
                            }
                            else
                            {
                                temp[threadIdx.x] = 0;
                            }
                        }
                    }
                }
            }

            if (input[i] > (15 + offset[6]))
            {
                if (input[i] > (23 + offset[6]))
                {
                    if (input[i] > (27 + offset[6]))
                    {
                        if (input[i] > (29 + offset[6]))
                        {
                            if (input[i] == (31 + offset[6]))
                            {
                                temp[threadIdx.x] = 31;
                            }
                            else
                            {
                                temp[threadIdx.x] = 30;
                            }
                        }
                        else
                        {
                            if (input[i] == (29 + offset[6]))
                            {
                                temp[threadIdx.x] = 29;
                            }
                            else
                            {
                                temp[threadIdx.x] = 28;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (25 + offset[6]))
                        {
                            if (input[i] == (27 + offset[6]))
                            {
                                temp[threadIdx.x] = 27;
                            }
                            else
                            {
                                temp[threadIdx.x] = 26;
                            }
                        }
                        else
                        {
                            if (input[i] == (25 + offset[6]))
                            {
                                temp[threadIdx.x] = 25;
                            }
                            else
                            {
                                temp[threadIdx.x] = 24;
                            }
                        }
                    }
                }
                else
                {
                    if (input[i] > (19 + offset[6]))
                    {
                        if (input[i] > (21 + offset[6]))
                        {
                            if (input[i] == (23 + offset[6]))
                            {
                                temp[threadIdx.x] = 23;
                            }
                            else
                            {
                                temp[threadIdx.x] = 22;
                            }
                        }
                        else
                        {
                            if (input[i] == (21 + offset[6]))
                            {
                                temp[threadIdx.x] = 21;
                            }
                            else
                            {
                                temp[threadIdx.x] = 20;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (17 + offset[6]))
                        {
                            if (input[i] == (19 + offset[6]))
                            {
                                temp[threadIdx.x] = 19;
                            }
                            else
                            {
                                temp[threadIdx.x] = 18;
                            }
                        }
                        else
                        {
                            if (input[i] == (17 + offset[6]))
                            {
                                temp[threadIdx.x] = 17;
                            }
                            else
                            {
                                temp[threadIdx.x] = 16;
                            }
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (7 + offset[6]))
                {
                    if (input[i] > (11 + offset[6]))
                    {
                        if (input[i] > (13 + offset[6]))
                        {
                            if (input[i] == (15 + offset[6]))
                            {
                                temp[threadIdx.x] = 15;
                            }
                            else
                            {
                                temp[threadIdx.x] = 14;
                            }
                        }
                        else
                        {
                            if (input[i] == (13 + offset[6]))
                            {
                                temp[threadIdx.x] = 13;
                            }
                            else
                            {
                                temp[threadIdx.x] = 12;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (9 + offset[6]))
                        {
                            if (input[i] == (11 + offset[6]))
                            {
                                temp[threadIdx.x] = 11;
                            }
                            else
                            {
                                temp[threadIdx.x] = 10;
                            }
                        }
                        else
                        {
                            if (input[i] == (9 + offset[6]))
                            {
                                temp[threadIdx.x] = 9;
                            }
                            else
                            {
                                temp[threadIdx.x] = 8;
                            }
                        }
                    }
                }
                else
                {
                    if (input[i] > (3 + offset[6]))
                    {
                        if (input[i] > (5 + offset[6]))
                        {
                            if (input[i] == (7 + offset[6]))
                            {
                                temp[threadIdx.x] = 7;
                            }
                            else
                            {
                                temp[threadIdx.x] = 6;
                            }
                        }
                        else
                        {
                            if (input[i] == (5 + offset[6]))
                            {
                                temp[threadIdx.x] = 5;
                            }
                            else
                            {
                                temp[threadIdx.x] = 4;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (1 + offset[6]))
                        {
                            if (input[i] == (3 + offset[6]))
                            {
                                temp[threadIdx.x] = 3;
                            }
                            else
                            {
                                temp[threadIdx.x] = 2;
                            }
                        }
                        else
                        {
                            if (input[i] == (1 + offset[6]))
                            {
                                temp[threadIdx.x] = 1;
                            }
                            else
                            {
                                temp[threadIdx.x] = 0;
                            }
                        }
                    }
                }
            }

            if (input[i] > (15 + offset[7]))
            {
                if (input[i] > (23 + offset[7]))
                {
                    if (input[i] > (27 + offset[7]))
                    {
                        if (input[i] > (29 + offset[7]))
                        {
                            if (input[i] == (31 + offset[7]))
                            {
                                temp[threadIdx.x] = 31;
                            }
                            else
                            {
                                temp[threadIdx.x] = 30;
                            }
                        }
                        else
                        {
                            if (input[i] == (29 + offset[7]))
                            {
                                temp[threadIdx.x] = 29;
                            }
                            else
                            {
                                temp[threadIdx.x] = 28;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (25 + offset[7]))
                        {
                            if (input[i] == (27 + offset[7]))
                            {
                                temp[threadIdx.x] = 27;
                            }
                            else
                            {
                                temp[threadIdx.x] = 26;
                            }
                        }
                        else
                        {
                            if (input[i] == (25 + offset[7]))
                            {
                                temp[threadIdx.x] = 25;
                            }
                            else
                            {
                                temp[threadIdx.x] = 24;
                            }
                        }
                    }
                }
                else
                {
                    if (input[i] > (19 + offset[7]))
                    {
                        if (input[i] > (21 + offset[7]))
                        {
                            if (input[i] == (23 + offset[7]))
                            {
                                temp[threadIdx.x] = 23;
                            }
                            else
                            {
                                temp[threadIdx.x] = 22;
                            }
                        }
                        else
                        {
                            if (input[i] == (21 + offset[7]))
                            {
                                temp[threadIdx.x] = 21;
                            }
                            else
                            {
                                temp[threadIdx.x] = 20;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (17 + offset[7]))
                        {
                            if (input[i] == (19 + offset[7]))
                            {
                                temp[threadIdx.x] = 19;
                            }
                            else
                            {
                                temp[threadIdx.x] = 18;
                            }
                        }
                        else
                        {
                            if (input[i] == (17 + offset[7]))
                            {
                                temp[threadIdx.x] = 17;
                            }
                            else
                            {
                                temp[threadIdx.x] = 16;
                            }
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (7 + offset[7]))
                {
                    if (input[i] > (11 + offset[7]))
                    {
                        if (input[i] > (13 + offset[7]))
                        {
                            if (input[i] == (15 + offset[7]))
                            {
                                temp[threadIdx.x] = 15;
                            }
                            else
                            {
                                temp[threadIdx.x] = 14;
                            }
                        }
                        else
                        {
                            if (input[i] == (13 + offset[7]))
                            {
                                temp[threadIdx.x] = 13;
                            }
                            else
                            {
                                temp[threadIdx.x] = 12;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (9 + offset[7]))
                        {
                            if (input[i] == (11 + offset[7]))
                            {
                                temp[threadIdx.x] = 11;
                            }
                            else
                            {
                                temp[threadIdx.x] = 10;
                            }
                        }
                        else
                        {
                            if (input[i] == (9 + offset[7]))
                            {
                                temp[threadIdx.x] = 9;
                            }
                            else
                            {
                                temp[threadIdx.x] = 8;
                            }
                        }
                    }
                }
                else
                {
                    if (input[i] > (3 + offset[7]))
                    {
                        if (input[i] > (5 + offset[7]))
                        {
                            if (input[i] == (7 + offset[7]))
                            {
                                temp[threadIdx.x] = 7;
                            }
                            else
                            {
                                temp[threadIdx.x] = 6;
                            }
                        }
                        else
                        {
                            if (input[i] == (5 + offset[7]))
                            {
                                temp[threadIdx.x] = 5;
                            }
                            else
                            {
                                temp[threadIdx.x] = 4;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (1 + offset[7]))
                        {
                            if (input[i] == (3 + offset[7]))
                            {
                                temp[threadIdx.x] = 3;
                            }
                            else
                            {
                                temp[threadIdx.x] = 2;
                            }
                        }
                        else
                        {
                            if (input[i] == (1 + offset[7]))
                            {
                                temp[threadIdx.x] = 1;
                            }
                            else
                            {
                                temp[threadIdx.x] = 0;
                            }
                        }
                    }
                }
            }

            if (input[i] > (15 + offset[8]))
            {
                if (input[i] > (23 + offset[8]))
                {
                    if (input[i] > (27 + offset[8]))
                    {
                        if (input[i] > (29 + offset[8]))
                        {
                            if (input[i] == (31 + offset[8]))
                            {
                                temp[threadIdx.x] = 31;
                            }
                            else
                            {
                                temp[threadIdx.x] = 30;
                            }
                        }
                        else
                        {
                            if (input[i] == (29 + offset[8]))
                            {
                                temp[threadIdx.x] = 29;
                            }
                            else
                            {
                                temp[threadIdx.x] = 28;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (25 + offset[8]))
                        {
                            if (input[i] == (27 + offset[8]))
                            {
                                temp[threadIdx.x] = 27;
                            }
                            else
                            {
                                temp[threadIdx.x] = 26;
                            }
                        }
                        else
                        {
                            if (input[i] == (25 + offset[8]))
                            {
                                temp[threadIdx.x] = 25;
                            }
                            else
                            {
                                temp[threadIdx.x] = 24;
                            }
                        }
                    }
                }
                else
                {
                    if (input[i] > (19 + offset[8]))
                    {
                        if (input[i] > (21 + offset[8]))
                        {
                            if (input[i] == (23 + offset[8]))
                            {
                                temp[threadIdx.x] = 23;
                            }
                            else
                            {
                                temp[threadIdx.x] = 22;
                            }
                        }
                        else
                        {
                            if (input[i] == (21 + offset[8]))
                            {
                                temp[threadIdx.x] = 21;
                            }
                            else
                            {
                                temp[threadIdx.x] = 20;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (17 + offset[8]))
                        {
                            if (input[i] == (19 + offset[8]))
                            {
                                temp[threadIdx.x] = 19;
                            }
                            else
                            {
                                temp[threadIdx.x] = 18;
                            }
                        }
                        else
                        {
                            if (input[i] == (17 + offset[8]))
                            {
                                temp[threadIdx.x] = 17;
                            }
                            else
                            {
                                temp[threadIdx.x] = 16;
                            }
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (7 + offset[8]))
                {
                    if (input[i] > (11 + offset[8]))
                    {
                        if (input[i] > (13 + offset[8]))
                        {
                            if (input[i] == (15 + offset[8]))
                            {
                                temp[threadIdx.x] = 15;
                            }
                            else
                            {
                                temp[threadIdx.x] = 14;
                            }
                        }
                        else
                        {
                            if (input[i] == (13 + offset[8]))
                            {
                                temp[threadIdx.x] = 13;
                            }
                            else
                            {
                                temp[threadIdx.x] = 12;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (9 + offset[8]))
                        {
                            if (input[i] == (11 + offset[8]))
                            {
                                temp[threadIdx.x] = 11;
                            }
                            else
                            {
                                temp[threadIdx.x] = 10;
                            }
                        }
                        else
                        {
                            if (input[i] == (9 + offset[8]))
                            {
                                temp[threadIdx.x] = 9;
                            }
                            else
                            {
                                temp[threadIdx.x] = 8;
                            }
                        }
                    }
                }
                else
                {
                    if (input[i] > (3 + offset[8]))
                    {
                        if (input[i] > (5 + offset[8]))
                        {
                            if (input[i] == (7 + offset[8]))
                            {
                                temp[threadIdx.x] = 7;
                            }
                            else
                            {
                                temp[threadIdx.x] = 6;
                            }
                        }
                        else
                        {
                            if (input[i] == (5 + offset[8]))
                            {
                                temp[threadIdx.x] = 5;
                            }
                            else
                            {
                                temp[threadIdx.x] = 4;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (1 + offset[8]))
                        {
                            if (input[i] == (3 + offset[8]))
                            {
                                temp[threadIdx.x] = 3;
                            }
                            else
                            {
                                temp[threadIdx.x] = 2;
                            }
                        }
                        else
                        {
                            if (input[i] == (1 + offset[8]))
                            {
                                temp[threadIdx.x] = 1;
                            }
                            else
                            {
                                temp[threadIdx.x] = 0;
                            }
                        }
                    }
                }
            }

            if (input[i] > (15 + offset[9]))
            {
                if (input[i] > (23 + offset[9]))
                {
                    if (input[i] > (27 + offset[9]))
                    {
                        if (input[i] > (29 + offset[9]))
                        {
                            if (input[i] == (31 + offset[9]))
                            {
                                temp[threadIdx.x] = 31;
                            }
                            else
                            {
                                temp[threadIdx.x] = 30;
                            }
                        }
                        else
                        {
                            if (input[i] == (29 + offset[9]))
                            {
                                temp[threadIdx.x] = 29;
                            }
                            else
                            {
                                temp[threadIdx.x] = 28;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (25 + offset[9]))
                        {
                            if (input[i] == (27 + offset[9]))
                            {
                                temp[threadIdx.x] = 27;
                            }
                            else
                            {
                                temp[threadIdx.x] = 26;
                            }
                        }
                        else
                        {
                            if (input[i] == (25 + offset[9]))
                            {
                                temp[threadIdx.x] = 25;
                            }
                            else
                            {
                                temp[threadIdx.x] = 24;
                            }
                        }
                    }
                }
                else
                {
                    if (input[i] > (19 + offset[9]))
                    {
                        if (input[i] > (21 + offset[9]))
                        {
                            if (input[i] == (23 + offset[9]))
                            {
                                temp[threadIdx.x] = 23;
                            }
                            else
                            {
                                temp[threadIdx.x] = 22;
                            }
                        }
                        else
                        {
                            if (input[i] == (21 + offset[9]))
                            {
                                temp[threadIdx.x] = 21;
                            }
                            else
                            {
                                temp[threadIdx.x] = 20;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (17 + offset[9]))
                        {
                            if (input[i] == (19 + offset[9]))
                            {
                                temp[threadIdx.x] = 19;
                            }
                            else
                            {
                                temp[threadIdx.x] = 18;
                            }
                        }
                        else
                        {
                            if (input[i] == (17 + offset[9]))
                            {
                                temp[threadIdx.x] = 17;
                            }
                            else
                            {
                                temp[threadIdx.x] = 16;
                            }
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (7 + offset[9]))
                {
                    if (input[i] > (11 + offset[9]))
                    {
                        if (input[i] > (13 + offset[9]))
                        {
                            if (input[i] == (15 + offset[9]))
                            {
                                temp[threadIdx.x] = 15;
                            }
                            else
                            {
                                temp[threadIdx.x] = 14;
                            }
                        }
                        else
                        {
                            if (input[i] == (13 + offset[9]))
                            {
                                temp[threadIdx.x] = 13;
                            }
                            else
                            {
                                temp[threadIdx.x] = 12;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (9 + offset[9]))
                        {
                            if (input[i] == (11 + offset[9]))
                            {
                                temp[threadIdx.x] = 11;
                            }
                            else
                            {
                                temp[threadIdx.x] = 10;
                            }
                        }
                        else
                        {
                            if (input[i] == (9 + offset[9]))
                            {
                                temp[threadIdx.x] = 9;
                            }
                            else
                            {
                                temp[threadIdx.x] = 8;
                            }
                        }
                    }
                }
                else
                {
                    if (input[i] > (3 + offset[9]))
                    {
                        if (input[i] > (5 + offset[9]))
                        {
                            if (input[i] == (7 + offset[9]))
                            {
                                temp[threadIdx.x] = 7;
                            }
                            else
                            {
                                temp[threadIdx.x] = 6;
                            }
                        }
                        else
                        {
                            if (input[i] == (5 + offset[9]))
                            {
                                temp[threadIdx.x] = 5;
                            }
                            else
                            {
                                temp[threadIdx.x] = 4;
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > (1 + offset[9]))
                        {
                            if (input[i] == (3 + offset[9]))
                            {
                                temp[threadIdx.x] = 3;
                            }
                            else
                            {
                                temp[threadIdx.x] = 2;
                            }
                        }
                        else
                        {
                            if (input[i] == (1 + offset[9]))
                            {
                                temp[threadIdx.x] = 1;
                            }
                            else
                            {
                                temp[threadIdx.x] = 0;
                            }
                        }
                    }
                }
            }
        }
    }

    __syncthreads();
    if (threadIdx.x < 512)
    {
        vR[i] = temp[threadIdx.x] + temp[threadIdx.x + 512];
    }
}

void initData(float *ip, int size);
void registerTime(float value);

int main(int argc, char **argv)
{
    float elapsed_time;
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("[CUDA - Random Forest]: EXP %s\n", EXP_NAME);
    printf("[CUDA - Random Forest]: Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set up data size of vectors
    int nElem = N_ELEM;

    printf("[CUDA - Random Forest]: Vector Size %d\n", nElem);

    // malloc host memory
    size_t nBytes = nElem * sizeof(float);
    size_t offsetBytes = OFFSET_SIZE * sizeof(int);

    float *h_vR;
    h_vR = (float *)malloc(nBytes);
    float *h_input;
    h_input = (float *)malloc(nBytes);
    int *h_offset = (int *)malloc(offsetBytes);

    printf("[CUDA - Random Forest]: Start Reading Dataset\n");
    initData(h_input, nElem);
    printf("[CUDA - Random Forest]: Finished Reading Dataset\n");
    memset(h_vR, 0, nBytes);
    memset(h_offset, 0, offsetBytes);

    // malloc device global memory
    float *d_vR;
    CHECK(hipMalloc((float **)&d_vR, nBytes));
    float *d_input;
    CHECK(hipMalloc((float **)&d_input, nBytes));
    int *d_offset;
    CHECK(hipMalloc((float **)&d_offset, offsetBytes));

    // transfer data from host to device
    CHECK(hipMemcpy(d_input, h_input, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_offset, h_offset, offsetBytes, hipMemcpyHostToDevice));

    // invoke kernel at host side
    int iLen = 1024;
    dim3 block(iLen);
    dim3 grid((nElem + (block.x / 2) - 1) / (block.x / 2));
    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    // record start event
    CHECK(hipEventRecord(start, 0));

    RF_kernel<<<grid, block>>>(d_input, nElem, d_offset, d_vR);

    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    hipDeviceSynchronize();

    // calculate elapsed time
    CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
    printf("[CUDA - Random Forest]: %s - execution time = %.6fms\n", EXP_NAME, elapsed_time);
    registerTime(elapsed_time);

    // copy kernel result back to host side
    CHECK(hipMemcpy(h_vR, d_vR, nBytes, hipMemcpyDeviceToHost));
    CHECK(hipGetLastError());

    // Verify the answers
    char flag = 1;
    for (int i = 0; i < nElem; i++)
    {
        if (i < WARP_SIZE)
        {
            printf("%.3f ", h_vR[i]);
        }
        if (2 * h_input[i] != h_vR[i])
        {
            flag = 0;
            // break;
        }
    }

    if (flag == 1)
    {
        printf("Return correct");
    }
    else
    {
        printf("Error!!");
    }
    printf("\n ");

    // free device memory
    hipFree(d_input);
    hipFree(d_vR);
    hipFree(d_offset);

    // free host memory
    free(h_input);
    free(h_vR);
    free(h_offset);

    return (0);
}

void initData(float *ip, int size)
{
    for (int i = 0; i < size; i++)
    {

        if (WARP_SIZE <= _MOD)
        {
            int aux = _MOD / WARP_SIZE;
            ip[i] = (float)((i * aux) % _MOD);
        }
        else
        {
            ip[i] = (float)(i % _MOD);
        }
        // if (i < 32)
        //	printf("%.2f ", ip[i]);
    }
    return;
}

void registerTime(float value)
{
    printf("[CUDA - Random Forest]: Time spent: %f.\n", value);
}
