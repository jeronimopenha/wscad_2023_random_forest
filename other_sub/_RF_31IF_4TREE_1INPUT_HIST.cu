#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>

#define TH_PER_BLOCK 512

#define _WORST
// #define _BEST
#define N_ELEM 50000000

#define WARP_SIZE 32

#define CHECK(call)                                                \
	{                                                              \
		const hipError_t error = call;                            \
		if (error != hipSuccess)                                  \
		{                                                          \
			fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
			fprintf(stderr, "code: %d, reason: %s\n", error,       \
					hipGetErrorString(error));                    \
		}                                                          \
	}

void initData(float *ip, int size);
void registerTime(float value);

#define _MOD 32
#define _4TREE
#define EXP_NAME "_RF_31IF_4TREE_1INPUT__WORST_50000000"

// RF_with_if

__global__ void RF_kernel(float *input, const int n, const int offset0, const int offset1, const int offset2, const int offset3, float *R, int *histogram)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int vR;

	__shared__ int th_idx[TH_PER_BLOCK];

	th_idx[threadIdx.x] = 0;

	if (i < n)
	{
		if (input[i] > (15 + offset0))
		{
			if (input[i] > (23 + offset0))
			{
				if (input[i] > (27 + offset0))
				{
					if (input[i] > (29 + offset0))
					{
						if (input[i] == (31 + offset0))
						{
							vR = 31;
						}
						else
						{
							vR = 30;
						}
					}
					else
					{
						if (input[i] == (29 + offset0))
						{
							vR = 29;
						}
						else
						{
							vR = 28;
						}
					}
				}
				else
				{
					if (input[i] > (25 + offset0))
					{
						if (input[i] == (27 + offset0))
						{
							vR = 27;
						}
						else
						{
							vR = 26;
						}
					}
					else
					{
						if (input[i] == (25 + offset0))
						{
							vR = 25;
						}
						else
						{
							vR = 24;
						}
					}
				}
			}
			else
			{
				if (input[i] > (19 + offset0))
				{
					if (input[i] > (21 + offset0))
					{
						if (input[i] == (23 + offset0))
						{
							vR = 23;
						}
						else
						{
							vR = 22;
						}
					}
					else
					{
						if (input[i] == (21 + offset0))
						{
							vR = 21;
						}
						else
						{
							vR = 20;
						}
					}
				}
				else
				{
					if (input[i] > (17 + offset0))
					{
						if (input[i] == (19 + offset0))
						{
							vR = 19;
						}
						else
						{
							vR = 18;
						}
					}
					else
					{
						if (input[i] == (17 + offset0))
						{
							vR = 17;
						}
						else
						{
							vR = 16;
						}
					}
				}
			}
		}
		else
		{
			if (input[i] > (7 + offset0))
			{
				if (input[i] > (11 + offset0))
				{
					if (input[i] > (13 + offset0))
					{
						if (input[i] == (15 + offset0))
						{
							vR = 15;
						}
						else
						{
							vR = 14;
						}
					}
					else
					{
						if (input[i] == (13 + offset0))
						{
							vR = 13;
						}
						else
						{
							vR = 12;
						}
					}
				}
				else
				{
					if (input[i] > (9 + offset0))
					{
						if (input[i] == (11 + offset0))
						{
							vR = 11;
						}
						else
						{
							vR = 10;
						}
					}
					else
					{
						if (input[i] == (9 + offset0))
						{
							vR = 9;
						}
						else
						{
							vR = 8;
						}
					}
				}
			}
			else
			{
				if (input[i] > (3 + offset0))
				{
					if (input[i] > (5 + offset0))
					{
						if (input[i] == (7 + offset0))
						{
							vR = 7;
						}
						else
						{
							vR = 6;
						}
					}
					else
					{
						if (input[i] == (5 + offset0))
						{
							vR = 5;
						}
						else
						{
							vR = 4;
						}
					}
				}
				else
				{
					if (input[i] > (1 + offset0))
					{
						if (input[i] == (3 + offset0))
						{
							vR = 3;
						}
						else
						{
							vR = 2;
						}
					}
					else
					{
						if (input[i] == (1 + offset0))
						{
							vR = 1;
						}
						else
						{
							vR = 0;
						}
					}
				}
			}
		}

		R[i] = vR;
		th_idx[threadIdx.x] = vR;

		if (threadIdx.x % WARP_SIZE == 0)
		{
			int uniqueValues[WARP_SIZE];
			int count = 0;

			for (int i = 0; i < WARP_SIZE; i++)
			{
				int isUnique = 1;
				for (int j = 0; j < count; j++)
				{
					if (th_idx[threadIdx.x + i] == uniqueValues[j])
					{
						isUnique = 0;
						break;
					}
				}
				if (isUnique)
				{
					uniqueValues[count] = th_idx[threadIdx.x + i];
					count++;
				}
			}
			atomicAdd(&histogram[count-1], 1);
		}
	}
}

int main(int argc, char **argv)
{
	float elapsed_time;
	// set up device
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("[CUDA - Random Forest]: EXP %s\n", EXP_NAME);
	printf("[CUDA - Random Forest]: Using Device %d: %s\n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));

	// set up data size of vectors
	int nElem = N_ELEM;
	printf("[CUDA - Random Forest]: Vector Size %d\n", nElem);

	// malloc host memory
	size_t nBytes = nElem * sizeof(float);
	size_t histBytes = WARP_SIZE * sizeof(int);
	float *h_vR;
	h_vR = (float *)malloc(nBytes);
	float *h_input;
	h_input = (float *)malloc(nBytes);
	int *h_histogram;
	h_histogram = (int *)malloc(histBytes);

	printf("[CUDA - Random Forest]: Start Reading Dataset\n");
	initData(h_input, nElem);
	printf("[CUDA - Random Forest]: Finished Reading Dataset\n");
	memset(h_vR, 0, nBytes);
	memset(h_histogram, 0, histBytes);

	// malloc device global memory
	float *d_vR;
	CHECK(hipMalloc((float **)&d_vR, nBytes));
	float *d_input;
	CHECK(hipMalloc((float **)&d_input, nBytes));
	int *d_histogram;
	CHECK(hipMalloc((int **)&d_histogram, histBytes));

	// transfer data from host to device
	CHECK(hipMemcpy(d_input, h_input, nBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_histogram, h_histogram, histBytes, hipMemcpyHostToDevice));

	// invoke kernel at host side
	int iLen = TH_PER_BLOCK;
	dim3 block(iLen);
	dim3 grid((nElem + block.x - 1) / block.x);
	hipEvent_t start, stop;
	CHECK(hipEventCreate(&start));
	CHECK(hipEventCreate(&stop));

	// record start event
	CHECK(hipEventRecord(start, 0));

	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, d_vR, d_histogram);

	CHECK(hipEventRecord(stop, 0));
	CHECK(hipEventSynchronize(stop));
	hipDeviceSynchronize();

	// calculate elapsed time
	CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
	printf("[CUDA - Random Forest]: %s - execution time = %.6fms\n", EXP_NAME, elapsed_time);
	registerTime(elapsed_time);

	// copy kernel result back to host side
	CHECK(hipMemcpy(h_vR, d_vR, nBytes, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(h_histogram, d_histogram, histBytes, hipMemcpyDeviceToHost));
	CHECK(hipGetLastError());

	for (int i = 0; i < WARP_SIZE; i++)
	{
		printf("%d ", h_histogram[i]);
	}

	printf("\n\n");
	// Verify the answers
	char flag = 1;
	for (int i = 0; i < nElem; i++)
	{
		if (i < 32)
		{
			printf("%.3f ", h_vR[i]);
		}
		if (h_input[i] != h_vR[i])
		{
			flag = 0;
			break;
		}
	}
	if (flag == 1)
	{
		printf("Return correct");
	}
	else
	{
		printf("Error!!");
	}
	printf("\n ");

	// free device memory
	hipFree(d_input);
	hipFree(d_vR);
	hipFree(d_histogram);

	// free host memory
	free(h_input);
	free(h_vR);
	free(h_histogram);
	return (0);
}

void initData(float *ip, int size)
{
	for (int i = 0; i < size; i++)
	{

#if defined(_BEST)
		ip[i] = (float)(0);
#else
		if (WARP_SIZE < _MOD)
		{
			int aux = _MOD / WARP_SIZE;
			ip[i] = (float)((i * aux) % _MOD);
		}
		else
		{
			ip[i] = (float)(i % _MOD);
		}
#endif
	}
	return;
}

void registerTime(float value)
{
	printf("[CUDA - Random Forest]: Time spent: %f.\n", value);
}
