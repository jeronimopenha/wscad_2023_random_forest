#include "hip/hip_runtime.h"
#include <sys/time.h>
#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
    }                                                                          \
}
#define N_ELEM 4999998


#include <hip/hip_runtime.h>
#include <stdio.h>


void initialData(float *ip, int size){

  for (int i = 0; i < size; i++){
      ip[i] = (float)( rand() & 15 );
  }
  return;
}
FILE *inFile;
FILE *outFile;

void readInFile(float **ip) {
    FILE *file = fopen("SUSY.csv","r");
    if (file == NULL) {
        printf("[CUDA]: Failed to open the file.\n");
        exit(1);
    }

    char line[1000];
    
    int row = 0;
    fgets(line, sizeof(line), file);
    while (fgets(line, sizeof(line), file)) {
        char *token = strtok(line, ",");
        
        int column = 0;
        while (token != NULL) {
            ip[column][row] = atof(token);
            token = strtok(NULL, ",");
            column++;
        }

        row++;
        if(row > N_ELEM) break;
    }

    fclose(file);
}
void writeOutFile(int value){
    outFile = fopen("out_rf.csv","a");
    fprintf(outFile, "%d\n", value);
    fclose(outFile);
} 

void registerTime(float value){
    outFile = fopen("results/results.csv","a");
    fprintf(outFile, ",%.4f", value);
    fclose(outFile);
}__global__ void RF_with_IF(float *F0, float *F1, float *F2, float *F3, float *F4, float *F5, float *F6, float *F7, float *F8, float *F9, float *F10, float *F11, float *F12, float *F13, float *F14, float *F15, float *F16, float *F17, float *F18, int *P, const int N, int offset)
{	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int Class[2]; 
	Class[0] = 0;
	Class[1] = 0;
	if (i < N) {
		int leaf;
		if (F10[i]<= (1.013+ offset)) {
			if (F15[i]<= (0.761+ offset)) {
				if (F7[i]<= (1.279+ offset)) {
					if (F15[i]<= (0.575+ offset)) {
						if (F1[i]<= (0.716+ offset)) {
							leaf = 0;
							Class[1]++;
						} else {
							leaf = 1;
							Class[0]++;
						}
					} else {
						if (F1[i]<= (0.824+ offset)) {
							leaf = 2;
							Class[1]++;
						} else {
							leaf = 3;
							Class[0]++;
						}
					}
				} else {
					if (F7[i]<= (1.633+ offset)) {
						if (F13[i]<= (1.718+ offset)) {
							leaf = 4;
							Class[0]++;
						} else {
							leaf = 5;
							Class[1]++;
						}
					} else {
						if (F10[i]<= (-0.851+ offset)) {
							leaf = 6;
							Class[0]++;
						} else {
							leaf = 7;
							Class[0]++;
						}
					}
				}
			} else {
				if (F7[i]<= (1.156+ offset)) {
					if (F7[i]<= (0.756+ offset)) {
						if (F0[i]<= (0.156+ offset)) {
							leaf = 8;
							Class[1]++;
						} else {
							leaf = 9;
							Class[0]++;
						}
					} else {
						if (F11[i]<= (0.922+ offset)) {
							leaf = 10;
							Class[1]++;
						} else {
							leaf = 11;
							Class[0]++;
						}
					}
				} else {
					if (F1[i]<= (1.005+ offset)) {
						if (F7[i]<= (1.57+ offset)) {
							leaf = 12;
							Class[0]++;
						} else {
							leaf = 13;
							Class[0]++;
						}
					} else {
						if (F13[i]<= (0.699+ offset)) {
							leaf = 14;
							Class[0]++;
						} else {
							leaf = 15;
							Class[0]++;
						}
					}
				}
			}
		} else {
			if (F7[i]<= (1.087+ offset)) {
				if (F17[i]<= (1.312+ offset)) {
					if (F1[i]<= (0.627+ offset)) {
						if (F2[i]<= (-1.216+ offset)) {
							leaf = 16;
							Class[0]++;
						} else {
							leaf = 17;
							Class[0]++;
						}
					} else {
						if (F12[i]<= (0.481+ offset)) {
							leaf = 18;
							Class[0]++;
						} else {
							leaf = 19;
							Class[0]++;
						}
					}
				} else {
					if (F4[i]<= (1.032+ offset)) {
						if (F11[i]<= (0.507+ offset)) {
							leaf = 20;
							Class[1]++;
						} else {
							leaf = 21;
							Class[1]++;
						}
					} else {
						if (F17[i]<= (1.466+ offset)) {
							leaf = 22;
							Class[0]++;
						} else {
							leaf = 23;
							Class[0]++;
						}
					}
				}
			} else {
				if (F7[i]<= (1.387+ offset)) {
					if (F11[i]<= (0.68+ offset)) {
						if (F17[i]<= (1.345+ offset)) {
							leaf = 24;
							Class[0]++;
						} else {
							leaf = 25;
							Class[0]++;
						}
					} else {
						if (F2[i]<= (1.573+ offset)) {
							leaf = 26;
							Class[0]++;
						} else {
							leaf = 27;
							Class[0]++;
						}
					}
				} else {
					if (F1[i]<= (0.723+ offset)) {
						if (F7[i]<= (1.578+ offset)) {
							leaf = 28;
							Class[0]++;
						} else {
							leaf = 29;
							Class[0]++;
						}
					} else {
						if (F4[i]<= (3.169+ offset)) {
							leaf = 30;
							Class[0]++;
						} else {
							leaf = 31;
							Class[0]++;
						}
					}
				}
			}
		}

		if (F7[i]<= (1.199+ offset)) {
			if (F15[i]<= (0.737+ offset)) {
				if (F10[i]<= (0.785+ offset)) {
					if (F1[i]<= (0.75+ offset)) {
						if (F16[i]<= (1.516+ offset)) {
							leaf = 0;
							Class[1]++;
						} else {
							leaf = 1;
							Class[1]++;
						}
					} else {
						if (F0[i]<= (0.443+ offset)) {
							leaf = 2;
							Class[1]++;
						} else {
							leaf = 3;
							Class[0]++;
						}
					}
				} else {
					if (F2[i]<= (-1.17+ offset)) {
						if (F14[i]<= (0.425+ offset)) {
							leaf = 4;
							Class[1]++;
						} else {
							leaf = 5;
							Class[0]++;
						}
					} else {
						if (F13[i]<= (0.679+ offset)) {
							leaf = 6;
							Class[0]++;
						} else {
							leaf = 7;
							Class[1]++;
						}
					}
				}
			} else {
				if (F1[i]<= (1.03+ offset)) {
					if (F10[i]<= (0.845+ offset)) {
						if (F2[i]<= (1.467+ offset)) {
							leaf = 8;
							Class[1]++;
						} else {
							leaf = 9;
							Class[1]++;
						}
					} else {
						if (F14[i]<= (0.429+ offset)) {
							leaf = 10;
							Class[0]++;
						} else {
							leaf = 11;
							Class[0]++;
						}
					}
				} else {
					if (F13[i]<= (0.552+ offset)) {
						if (F2[i]<= (-1.33+ offset)) {
							leaf = 12;
							Class[1]++;
						} else {
							leaf = 13;
							Class[0]++;
						}
					} else {
						if (F1[i]<= (1.382+ offset)) {
							leaf = 14;
							Class[0]++;
						} else {
							leaf = 15;
							Class[0]++;
						}
					}
				}
			}
		} else {
			if (F1[i]<= (0.777+ offset)) {
				if (F14[i]<= (1.388+ offset)) {
					if (F10[i]<= (0.445+ offset)) {
						if (F13[i]<= (2.119+ offset)) {
							leaf = 16;
							Class[0]++;
						} else {
							leaf = 17;
							Class[0]++;
						}
					} else {
						if (F7[i]<= (1.52+ offset)) {
							leaf = 18;
							Class[0]++;
						} else {
							leaf = 19;
							Class[0]++;
						}
					}
				} else {
					if (F12[i]<= (1.369+ offset)) {
						if (F10[i]<= (-0.427+ offset)) {
							leaf = 20;
							Class[1]++;
						} else {
							leaf = 21;
							Class[0]++;
						}
					} else {
						if (F15[i]<= (0.794+ offset)) {
							leaf = 22;
							Class[0]++;
						} else {
							leaf = 23;
							Class[0]++;
						}
					}
				}
			} else {
				if (F2[i]<= (1.405+ offset)) {
					if (F9[i]<= (2.554+ offset)) {
						if (F5[i]<= (-1.281+ offset)) {
							leaf = 24;
							Class[0]++;
						} else {
							leaf = 25;
							Class[0]++;
						}
					} else {
						if (F4[i]<= (0.759+ offset)) {
							leaf = 26;
							Class[0]++;
						} else {
							leaf = 27;
							Class[0]++;
						}
					}
				} else {
					if (F14[i]<= (0.102+ offset)) {
						if (F4[i]<= (3.355+ offset)) {
							leaf = 28;
							Class[0]++;
						} else {
							leaf = 29;
							Class[0]++;
						}
					} else {
						if (F10[i]<= (-0.348+ offset)) {
							leaf = 30;
							Class[0]++;
						} else {
							leaf = 31;
							Class[0]++;
						}
					}
				}
			}
		}

		if (F0[i]<= (0.3+ offset)) {
			if (F9[i]<= (1.864+ offset)) {
				if (F7[i]<= (1.242+ offset)) {
					if (F12[i]<= (1.048+ offset)) {
						if (F1[i]<= (0.846+ offset)) {
							leaf = 0;
							Class[1]++;
						} else {
							leaf = 1;
							Class[1]++;
						}
					} else {
						if (F12[i]<= (1.246+ offset)) {
							leaf = 2;
							Class[1]++;
						} else {
							leaf = 3;
							Class[0]++;
						}
					}
				} else {
					if (F16[i]<= (1.147+ offset)) {
						if (F16[i]<= (0.817+ offset)) {
							leaf = 4;
							Class[0]++;
						} else {
							leaf = 5;
							Class[0]++;
						}
					} else {
						if (F11[i]<= (4.207+ offset)) {
							leaf = 6;
							Class[0]++;
						} else {
							leaf = 7;
							Class[0]++;
						}
					}
				}
			} else {
				if (F12[i]<= (1.419+ offset)) {
					if (F16[i]<= (1.411+ offset)) {
						if (F13[i]<= (1.497+ offset)) {
							leaf = 8;
							Class[0]++;
						} else {
							leaf = 9;
							Class[0]++;
						}
					} else {
						if (F16[i]<= (2.21+ offset)) {
							leaf = 10;
							Class[0]++;
						} else {
							leaf = 11;
							Class[0]++;
						}
					}
				} else {
					if (F9[i]<= (2.614+ offset)) {
						if (F7[i]<= (1.702+ offset)) {
							leaf = 12;
							Class[0]++;
						} else {
							leaf = 13;
							Class[0]++;
						}
					} else {
						if (F1[i]<= (0.949+ offset)) {
							leaf = 14;
							Class[0]++;
						} else {
							leaf = 15;
							Class[0]++;
						}
					}
				}
			}
		} else {
			if (F7[i]<= (1.279+ offset)) {
				if (F12[i]<= (1.115+ offset)) {
					if (F1[i]<= (0.846+ offset)) {
						if (F11[i]<= (0.552+ offset)) {
							leaf = 16;
							Class[1]++;
						} else {
							leaf = 17;
							Class[1]++;
						}
					} else {
						if (F1[i]<= (1.103+ offset)) {
							leaf = 18;
							Class[0]++;
						} else {
							leaf = 19;
							Class[0]++;
						}
					}
				} else {
					if (F11[i]<= (0.759+ offset)) {
						if (F10[i]<= (-0.577+ offset)) {
							leaf = 20;
							Class[1]++;
						} else {
							leaf = 21;
							Class[0]++;
						}
					} else {
						if (F0[i]<= (0.459+ offset)) {
							leaf = 22;
							Class[0]++;
						} else {
							leaf = 23;
							Class[0]++;
						}
					}
				}
			} else {
				if (F9[i]<= (2.66+ offset)) {
					if (F11[i]<= (0.668+ offset)) {
						if (F14[i]<= (1.677+ offset)) {
							leaf = 24;
							Class[0]++;
						} else {
							leaf = 25;
							Class[0]++;
						}
					} else {
						if (F17[i]<= (1.142+ offset)) {
							leaf = 26;
							Class[0]++;
						} else {
							leaf = 27;
							Class[0]++;
						}
					}
				} else {
					if (F15[i]<= (0.898+ offset)) {
						if (F14[i]<= (2.384+ offset)) {
							leaf = 28;
							Class[0]++;
						} else {
							leaf = 29;
							Class[0]++;
						}
					} else {
						if (F10[i]<= (-1.218+ offset)) {
							leaf = 30;
							Class[0]++;
						} else {
							leaf = 31;
							Class[0]++;
						}
					}
				}
			}
		}

		if (F9[i]<= (1.879+ offset)) {
			if (F11[i]<= (0.755+ offset)) {
				if (F12[i]<= (1.085+ offset)) {
					if (F10[i]<= (1.031+ offset)) {
						if (F1[i]<= (0.723+ offset)) {
							leaf = 0;
							Class[1]++;
						} else {
							leaf = 1;
							Class[1]++;
						}
					} else {
						if (F14[i]<= (0.225+ offset)) {
							leaf = 2;
							Class[0]++;
						} else {
							leaf = 3;
							Class[0]++;
						}
					}
				} else {
					if (F9[i]<= (1.328+ offset)) {
						if (F7[i]<= (1.358+ offset)) {
							leaf = 4;
							Class[0]++;
						} else {
							leaf = 5;
							Class[0]++;
						}
					} else {
						if (F10[i]<= (-0.393+ offset)) {
							leaf = 6;
							Class[1]++;
						} else {
							leaf = 7;
							Class[0]++;
						}
					}
				}
			} else {
				if (F11[i]<= (1.04+ offset)) {
					if (F2[i]<= (1.372+ offset)) {
						if (F13[i]<= (1.071+ offset)) {
							leaf = 8;
							Class[1]++;
						} else {
							leaf = 9;
							Class[0]++;
						}
					} else {
						if (F7[i]<= (1.252+ offset)) {
							leaf = 10;
							Class[1]++;
						} else {
							leaf = 11;
							Class[0]++;
						}
					}
				} else {
					if (F12[i]<= (1.204+ offset)) {
						if (F7[i]<= (1.155+ offset)) {
							leaf = 12;
							Class[1]++;
						} else {
							leaf = 13;
							Class[0]++;
						}
					} else {
						if (F15[i]<= (3.318+ offset)) {
							leaf = 14;
							Class[0]++;
						} else {
							leaf = 15;
							Class[0]++;
						}
					}
				}
			}
		} else {
			if (F12[i]<= (1.436+ offset)) {
				if (F10[i]<= (-0.428+ offset)) {
					if (F9[i]<= (2.473+ offset)) {
						if (F12[i]<= (1.316+ offset)) {
							leaf = 16;
							Class[1]++;
						} else {
							leaf = 17;
							Class[0]++;
						}
					} else {
						if (F5[i]<= (-1.274+ offset)) {
							leaf = 18;
							Class[0]++;
						} else {
							leaf = 19;
							Class[0]++;
						}
					}
				} else {
					if (F9[i]<= (2.177+ offset)) {
						if (F7[i]<= (1.493+ offset)) {
							leaf = 20;
							Class[0]++;
						} else {
							leaf = 21;
							Class[0]++;
						}
					} else {
						if (F7[i]<= (1.71+ offset)) {
							leaf = 22;
							Class[0]++;
						} else {
							leaf = 23;
							Class[0]++;
						}
					}
				}
			} else {
				if (F14[i]<= (1.216+ offset)) {
					if (F7[i]<= (1.86+ offset)) {
						if (F11[i]<= (6.6+ offset)) {
							leaf = 24;
							Class[0]++;
						} else {
							leaf = 25;
							Class[1]++;
						}
					} else {
						if (F9[i]<= (2.906+ offset)) {
							leaf = 26;
							Class[0]++;
						} else {
							leaf = 27;
							Class[0]++;
						}
					}
				} else {
					if (F9[i]<= (2.597+ offset)) {
						if (F1[i]<= (1.393+ offset)) {
							leaf = 28;
							Class[0]++;
						} else {
							leaf = 29;
							Class[0]++;
						}
					} else {
						if (F10[i]<= (-1.031+ offset)) {
							leaf = 30;
							Class[0]++;
						} else {
							leaf = 31;
							Class[0]++;
						}
					}
				}
			}
		}
		int p0 = (Class[0] > Class[1])?0:1;
		int Q0 = (Class[0] > Class[1])?Class[0]:Class[1];
		P[i] = p0;
	}
}int main(int argc, char ** argv) {
     
    float elapsed_time;
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("[CUDA]: Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set up data size of vectors
    int nElem = 4999998;
    printf("[CUDA]: Vector Size %d\n", nElem);

    // malloc host memory
    size_t nBytes = nElem * sizeof(float);

    float  * h_0,  * h_1,  * h_2,  * h_3,  * h_4,  * h_5,  * h_6,  * h_7,  * h_8,  * h_9,  * h_10,  * h_11,  * h_12,  * h_13,  * h_14,  * h_15,  * h_16,  * h_17,  * h_18;
    int * h_P, * hostRef;
    	h_0 = (float * ) malloc(nBytes);
	h_1 = (float * ) malloc(nBytes);
	h_2 = (float * ) malloc(nBytes);
	h_3 = (float * ) malloc(nBytes);
	h_4 = (float * ) malloc(nBytes);
	h_5 = (float * ) malloc(nBytes);
	h_6 = (float * ) malloc(nBytes);
	h_7 = (float * ) malloc(nBytes);
	h_8 = (float * ) malloc(nBytes);
	h_9 = (float * ) malloc(nBytes);
	h_10 = (float * ) malloc(nBytes);
	h_11 = (float * ) malloc(nBytes);
	h_12 = (float * ) malloc(nBytes);
	h_13 = (float * ) malloc(nBytes);
	h_14 = (float * ) malloc(nBytes);
	h_15 = (float * ) malloc(nBytes);
	h_16 = (float * ) malloc(nBytes);
	h_17 = (float * ) malloc(nBytes);
	h_18 = (float * ) malloc(nBytes);
    hostRef = (int * ) malloc(nBytes);
    h_P = (int * ) malloc(nBytes);

    printf("[CUDA]: Start Reading Dataset\n");
	float* h[19] = { h_0, h_1, h_2, h_3, h_4, h_5, h_6, h_7, h_8, h_9, h_10, h_11, h_12, h_13, h_14, h_15, h_16, h_17, h_18};
	readInFile(h);
	printf("[CUDA]: Finished Reading Dataset\n");


    memset(hostRef, 0, nBytes);
    memset(h_P, 0, nBytes);

    // malloc device global memory
    float * d_0, * d_1, * d_2, * d_3, * d_4, * d_5, * d_6, * d_7, * d_8, * d_9, * d_10, * d_11, * d_12, * d_13, * d_14, * d_15, * d_16, * d_17, * d_18;
    int * d_P;
        CHECK(hipMalloc((float ** ) & d_0, nBytes));
    CHECK(hipMalloc((float ** ) & d_1, nBytes));
    CHECK(hipMalloc((float ** ) & d_2, nBytes));
    CHECK(hipMalloc((float ** ) & d_3, nBytes));
    CHECK(hipMalloc((float ** ) & d_4, nBytes));
    CHECK(hipMalloc((float ** ) & d_5, nBytes));
    CHECK(hipMalloc((float ** ) & d_6, nBytes));
    CHECK(hipMalloc((float ** ) & d_7, nBytes));
    CHECK(hipMalloc((float ** ) & d_8, nBytes));
    CHECK(hipMalloc((float ** ) & d_9, nBytes));
    CHECK(hipMalloc((float ** ) & d_10, nBytes));
    CHECK(hipMalloc((float ** ) & d_11, nBytes));
    CHECK(hipMalloc((float ** ) & d_12, nBytes));
    CHECK(hipMalloc((float ** ) & d_13, nBytes));
    CHECK(hipMalloc((float ** ) & d_14, nBytes));
    CHECK(hipMalloc((float ** ) & d_15, nBytes));
    CHECK(hipMalloc((float ** ) & d_16, nBytes));
    CHECK(hipMalloc((float ** ) & d_17, nBytes));
    CHECK(hipMalloc((float ** ) & d_18, nBytes));
    CHECK(hipMalloc((int ** ) & d_P, nBytes));


    // transfer data from host to device
        CHECK(hipMemcpy(d_0, h_0, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_1, h_1, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_2, h_2, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_3, h_3, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_4, h_4, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_5, h_5, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_6, h_6, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_7, h_7, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_8, h_8, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_9, h_9, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_10, h_10, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_11, h_11, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_12, h_12, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_13, h_13, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_14, h_14, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_15, h_15, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_16, h_16, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_17, h_17, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_18, h_18, nBytes, hipMemcpyHostToDevice));
    // invoke kernel at host side
    int iLen = 512;
    dim3 block(iLen);
    dim3 grid((nElem + block.x - 1) / block.x);

    hipEvent_t start, stop;
    CHECK(hipEventCreate( & start));
    CHECK(hipEventCreate( & stop));
    // record start event
    CHECK(hipEventRecord(start, 0));
    RF_with_IF << < grid, block >>> (d_0, d_1, d_2, d_3, d_4, d_5, d_6, d_7, d_8, d_9, d_10, d_11, d_12, d_13, d_14, d_15, d_16, d_17, d_18, d_P, nElem, 0);
    hipDeviceSynchronize();
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    // calculate elapsed time
    CHECK(hipEventElapsedTime( & elapsed_time, start, stop));
    printf("[CUDA]: RF with IF - execution time = %.6fms\n", elapsed_time);

    registerTime(elapsed_time);
    CHECK(hipMemcpy(h_P, d_P, nBytes, hipMemcpyDeviceToHost));
    CHECK(hipEventCreate( & start));
    CHECK(hipEventCreate( & stop));
    CHECK(hipGetLastError());

    // copy kernel result back to host side
    CHECK(hipMemcpy(h_P, d_P, nBytes, hipMemcpyDeviceToHost));
    printf("\n ");

    // free host memory
        hipFree(d_0);
    hipFree(d_1);
    hipFree(d_2);
    hipFree(d_3);
    hipFree(d_4);
    hipFree(d_5);
    hipFree(d_6);
    hipFree(d_7);
    hipFree(d_8);
    hipFree(d_9);
    hipFree(d_10);
    hipFree(d_11);
    hipFree(d_12);
    hipFree(d_13);
    hipFree(d_14);
    hipFree(d_15);
    hipFree(d_16);
    hipFree(d_17);
    hipFree(d_18);
    hipFree(d_P);
        free(h_0);
    free(h_1);
    free(h_2);
    free(h_3);
    free(h_4);
    free(h_5);
    free(h_6);
    free(h_7);
    free(h_8);
    free(h_9);
    free(h_10);
    free(h_11);
    free(h_12);
    free(h_13);
    free(h_14);
    free(h_15);
    free(h_16);
    free(h_17);
    free(h_18);
    free(hostRef);

    return (0);
 }