
#include <hip/hip_runtime.h>
__global__ void RF_kernel(float *input, float *input1, float *input2, const float *TH, const int *table, const int n, const int offset0, float *vR)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int next = 0, left, right, idx;
    float th;
    int v[4];

    int tab[3] = {1, 2, 3};

    v[1] = 12;
    v[2] = 128;
    v[3] = 1;

    if (i < n)
    {
        v[0] = input[i];

        idx = table[next] & 0x3;
        left = (table[next] >> 2) & 0xff;
        right = (table[next] >> 10) & 0xff;
        th = TH[next];
        next = (v[idx] > th) ? left : right;

        idx = table[next] & 0x3;
        left = (table[next] >> 2) & 0xff;
        right = (table[next] >> 10) & 0xff;
        th = TH[next];
        next += (v[idx] > th) ? left : right;

        idx = table[next] & 0x3;
        left = (table[next] >> 2) & 0xff;
        right = (table[next] >> 10) & 0xff;
        th = TH[next];
        next += (v[idx] > th) ? left : right;

        vR[i] = next;
    }
}

__global__ void RF_kernel(float *input, const float *TH, const int *table, const int n, const int offset0, float *vR)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int next = 0, left, right, idx;
    float th;
    int v[4];

    v[1] = 12;
    v[2] = 128;
    v[3] = 1;

    if (i < n)
    {
        int v = input[i];

        // idx = table[next]&0x3;
        left = (table[next] >> 2) & 0xff;
        right = (table[next] >> 10) & 0xff;
        th = TH[next];
        next = (v > th) ? left : right;

        // idx = table[next]&0x3;
        left = (table[next] >> 2) & 0xff;
        right = (table[next] >> 10) & 0xff;
        th = TH[next];
        next = (v > th) ? left : right;

        // idx = table[next]&0x3;
        left = (table[next] >> 2) & 0xff;
        right = (table[next] >> 10) & 0xff;
        th = TH[next];
        next = (v > th) ? left : right;

        vR[i] = next;
    }
}

__global__ void RF_kernel(float *input, float *t, const int *table, const int n, const int offset0, float *vR)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int next = 0, left, right, idx;
    float th;
    int v[4];

    __shared__ float TH[15];
    if (threadIdx.x < 15)
    {
        TH[threadIdx.x] = t[threadIdx.x];
    }

    v[1] = 12;
    v[2] = 128;
    v[3] = 1;

    if (i < n)
    {
        int v = input[i];

        // idx = table[next]&0x3;
        left = (table[next] >> 2) & 0xff;
        right = (table[next] >> 10) & 0xff;
        th = TH[next];
        next = (v > th) ? left : right;

        // idx = table[next]&0x3;
        left = (table[next] >> 2) & 0xff;
        right = (table[next] >> 10) & 0xff;
        th = TH[next];
        next = (v > th) ? left : right;

        // idx = table[next]&0x3;
        left = (table[next] >> 2) & 0xff;
        right = (table[next] >> 10) & 0xff;
        th = TH[next];
        next = (v > th) ? left : right;

        vR[i] = next;
    }
}

__global__ void RF_kernel_div_warp(float *input, float *input1, float *input2, const float *TH, const int *table, const int n, const int offset0, float *vR)
{
    int i = blockIdx.x * blockDim.x/2 + threadIdx.x%512;
    int next = 0, left, right, idx;
    float th;
    __shared__ int temp[1024];

    //rodar os de prof 5 e 6 com 4 árvores
    if(threadIdx.x < 512){
        //2 árvores
    }else{
        //outras duas
    }

    if (i < n)
    {
        //
    }

    __syncthreads();
    //retornar
}
