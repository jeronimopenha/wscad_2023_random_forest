#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>

#define TAM_TABLE 15
#define TAM_TH 7

#define N_ELEM 50000000
// #define N_ELEM 100000000

#define INDIRECT

// #define _RF_TABLE_1TREE
// #define _RF_TABLE_2TREE_GLOBAL
// #define _RF_TABLE_3TREE_GLOBAL
// #define _RF_TABLE_4TREE_GLOBAL
// #define _RF_TABLE_1TREE_RESTRICT
// #define _RF_TABLE_2TREE_RESTRICT
// #define _RF_TABLE_3TREE_RESTRICT
// #define _RF_TABLE_4TREE_RESTRICT
// #define _RF_TABLE_1TREE_CONST
// #define _RF_TABLE_2TREE_CONST
// #define _RF_TABLE_3TREE_CONST
// #define _RF_TABLE_4TREE_CONST
#define _RF_TABLE_1TREE_SHARED
// #define _RF_TABLE_2TREE_SHARED
// #define _RF_TABLE_3TREE_SHARED
// #define _RF_TABLE_4TREE_SHARED

#define WARP_SIZE 32

#define CHECK(call)                                                \
	{                                                              \
		const hipError_t error = call;                            \
		if (error != hipSuccess)                                  \
		{                                                          \
			fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
			fprintf(stderr, "code: %d, reason: %s\n", error,       \
					hipGetErrorString(error));                    \
		}                                                          \
	}

#if defined(_RF_TABLE_1TREE_CONST) || defined(_RF_TABLE_2TREE_CONST) || \
	defined(_RF_TABLE_3TREE_CONST) || defined(_RF_TABLE_4TREE_CONST)
__constant__ int table[TAM_TABLE];
int tableData[] = {(1 << 2) | (2 << 10), (3 << 2) | (4 << 10), (5 << 2) | (6 << 10),
				   (7 << 2) | (8 << 10), (9 << 2) | (10 << 10), (11 << 2) | (12 << 10),
				   (13 << 2) | (14 << 10), 7, 6, 5, 4, 3, 2, 1, 0};
__constant__ float TH[TAM_TH];
float THData[] = {3, 5, 1, 6, 4, 2, 0};
#endif

#if defined(_RF_TABLE_1TREE_GLOBAL) || defined(_RF_TABLE_1TREE_RESTRICT) || defined(_RF_TABLE_1TREE_CONST) || defined(_RF_TABLE_1TREE_SHARED)
#define _MOD 4
#define _1TREE

#if defined(_RF_TABLE_1TREE_GLOBAL)
#define EXP_NAME "_RF_TABLE_1TREE_GLOBAL"
__global__ void
RF_kernel(float *input, const int n, const float *TH, const int *table, const int offset0, float *vR)
#elif defined(_RF_TABLE_1TREE_RESTRICT)
#define EXP_NAME "_RF_TABLE_1TREE_RESTRICT"
__global__ void
RF_kernel(float *input, const int n, float *__restrict__ TH, int *__restrict__ table, const int offset0, float *vR)
#elif defined(_RF_TABLE_1TREE_CONST)
#define EXP_NAME "_RF_TABLE_1TREE_CONST"
__global__ void RF_kernel(float *input, const int n, const int offset0, float *vR)
#elif defined(_RF_TABLE_1TREE_SHARED)
#define EXP_NAME "_RF_TABLE_1TREE_SHARED"
__global__ void RF_kernel(float *input, const int n, float *t, int *tab, const int offset0, float *vR)
#endif
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int next = 0, left, right;
	float th;

#ifdef _RF_TABLE_1TREE_SHARED
	__shared__ float TH[TAM_TH];
	__shared__ int table[TAM_TABLE];

	if (threadIdx.x < TAM_TH)
	{
		TH[threadIdx.x] = t[threadIdx.x];
	}
	if (threadIdx.x < TAM_TABLE)
	{
		table[threadIdx.x] = tab[threadIdx.x];
	}
	__syncthreads();
#endif

#ifdef INDIRECT
	int idx;
	int v[4];

	v[1] = 12;
	v[2] = 128;
	v[3] = 1;
#else
	int v;
#endif
	if (i < n)
	{

#ifdef INDIRECT
		v[0] = input[i];
#else
		v = input[i];
#endif

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset0)) ? left : right;
#else
		next = (v > (th + offset0)) ? left : right;
#endif

		// if (i == 2)
		//	printf("idx=%d, left=%d , right=%d , th=%f , next=%d \n", idx, left, right, th, next);

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset0)) ? left : right;
#else
		next = (v > (th + offset0)) ? left : right;
#endif

		// if (i == 2)
		//	printf("idx=%d, left=%d , right=%d , th=%f , next=%d \n", idx, left, right, th, next);

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset0)) ? left : right;
#else
		next = (v > (th + offset0)) ? left : right;
#endif

		// if (i == 2)
		//	printf("idx=%d, left=%d , right=%d , th=%f , next=%d \n", idx, left, right, th, next);

		vR[i] = table[next] + offset0;
	}
}
#endif

#if defined(_RF_TABLE_2TREE_GLOBAL) || defined(_RF_TABLE_2TREE_RESTRICT) || defined(_RF_TABLE_2TREE_CONST) || defined(_RF_TABLE_2TREE_SHARED)
#define _MOD 4
#define _2TREE

#if defined(_RF_TABLE_2TREE_GLOBAL)
#define EXP_NAME "_RF_TABLE_2TREE_GLOBAL"
__global__ void RF_kernel(float *input, const int n, const float *TH, const int *table, const int offset0, const int offset1, float *vR)
#elif defined(_RF_TABLE_2TREE_RESTRICT)
#define EXP_NAME "_RF_TABLE_2TREE_RESTRICT"
__global__ void RF_kernel(float *input, const int n, float *__restrict__ TH, int *__restrict__ table, const int offset0, const int offset1, float *vR)
#elif defined(_RF_TABLE_2TREE_CONST)
#define EXP_NAME "_RF_TABLE_2TREE_CONST"
__global__ void RF_kernel(float *input, const int n, const int offset0, const int offset1, float *vR)
#elif defined(_RF_TABLE_2TREE_SHARED)
#define EXP_NAME "_RF_TABLE_2TREE_SHARED"
__global__ void RF_kernel(float *input, const int n, const float *t, const int *tab, const int offset0, const int offset1, float *vR)
#endif
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int next = 0, left, right;
	float th;
#ifdef _RF_TABLE_2TREE_SHARED
	__shared__ float TH[TAM_TH];
	__shared__ int table[TAM_TABLE];

	if (threadIdx.x < TAM_TH)
	{
		TH[threadIdx.x] = t[threadIdx.x];
	}
	if (threadIdx.x < TAM_TABLE)
	{
		table[threadIdx.x] = tab[threadIdx.x];
	}
	__syncthreads();
#endif

#ifdef INDIRECT
	int idx;
	int v[4];

	v[1] = 12;
	v[2] = 128;
	v[3] = 1;
#else
	int v;
#endif
	if (i < n)
	{

#ifdef INDIRECT
		v[0] = input[i];
#else
		v = input[i];
#endif

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset0)) ? left : right;
#else
		next = (v > (th + offset0)) ? left : right;
#endif

		// if (i == 2)
		//	printf("idx=%d, left=%d , right=%d , th=%f , next=%d \n", idx, left, right, th, next);

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset0)) ? left : right;
#else
		next = (v > (th + offset0)) ? left : right;
#endif

		// if (i == 2)
		//	printf("idx=%d, left=%d , right=%d , th=%f , next=%d \n", idx, left, right, th, next);

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset0)) ? left : right;
#else
		next = (v > (th + offset0)) ? left : right;
#endif

		// if (i == 2)
		//	printf("idx=%d, left=%d , right=%d , th=%f , next=%d \n", idx, left, right, th, next);

		vR[i] = table[next] + offset0;

		// 2nd tree
#ifdef INDIRECT
		v[0] = input[i];
#else
		v = input[i];
#endif
		next = offset1;
#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset1)) ? left : right;
#else
		next = (v > (th + offset1)) ? left : right;
#endif

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset1)) ? left : right;
#else
		next = (v > (th + offset1)) ? left : right;
#endif

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset1)) ? left : right;
#else
		next = (v > (th + offset1)) ? left : right;
#endif

		vR[i] = table[next] + offset1;
	}
}
#endif

#if defined(_RF_TABLE_3TREE_GLOBAL) || defined(_RF_TABLE_3TREE_RESTRICT) || defined(_RF_TABLE_3TREE_CONST) || defined(_RF_TABLE_3TREE_SHARED)
#define _MOD 4
#define _3TREE

#if defined(_RF_TABLE_3TREE_GLOBAL)
#define EXP_NAME "_RF_TABLE_3TREE_GLOBAL"
__global__ void RF_kernel(float *input, const int n, const float *TH, const int *table, const int offset0, const int offset1, const int offset2, float *vR)
#elif defined(_RF_TABLE_3TREE_RESTRICT)
#define EXP_NAME "_RF_TABLE_3TREE_RESTRICT"
__global__ void RF_kernel(float *input, const int n, float *__restrict__ TH, int *__restrict__ table, const int offset0, const int offset1, const int offset2, float *vR)
#elif defined(_RF_TABLE_3TREE_CONST)
#define EXP_NAME "_RF_TABLE_3TREE_RESTRICT"
__global__ void RF_kernel(float *input, const int n, const int offset0, const int offset1, const int offset2, float *vR)
#elif defined(_RF_TABLE_3TREE_SHARED)
#define EXP_NAME "_RF_TABLE_3TREE_SHARED"
__global__ void RF_kernel(float *input, const int n, const float *t, const int *tab, const int offset0, const int offset1, const int offset2, float *vR)
#endif
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int next = 0, left, right;
	float th;
#ifdef _RF_TABLE_3TREE_SHARED
	__shared__ float TH[TAM_TH];
	__shared__ int table[TAM_TABLE];

	if (threadIdx.x < TAM_TH)
	{
		TH[threadIdx.x] = t[threadIdx.x];
	}
	if (threadIdx.x < TAM_TABLE)
	{
		table[threadIdx.x] = tab[threadIdx.x];
	}
	__syncthreads();
#endif

#ifdef INDIRECT
	int idx;
	int v[4];

	v[1] = 12;
	v[2] = 128;
	v[3] = 1;
#else
	int v;
#endif
	if (i < n)
	{

#ifdef INDIRECT
		v[0] = input[i];
#else
		v = input[i];
#endif

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset0)) ? left : right;
#else
		next = (v > (th + offset0)) ? left : right;
#endif

		// if (i == 2)
		//	printf("idx=%d, left=%d , right=%d , th=%f , next=%d \n", idx, left, right, th, next);

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset0)) ? left : right;
#else
		next = (v > (th + offset0)) ? left : right;
#endif

		// if (i == 2)
		//	printf("idx=%d, left=%d , right=%d , th=%f , next=%d \n", idx, left, right, th, next);

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset0)) ? left : right;
#else
		next = (v > (th + offset0)) ? left : right;
#endif

		// if (i == 2)
		//	printf("idx=%d, left=%d , right=%d , th=%f , next=%d \n", idx, left, right, th, next);

		vR[i] = table[next] + offset0;

// 2nd tree
#ifdef INDIRECT
		v[0] = input[i];
#else
		v = input[i];
#endif
		next = offset1;

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset1)) ? left : right;
#else
		next = (v > (th + offset1)) ? left : right;
#endif

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset1)) ? left : right;
#else
		next = (v > (th + offset1)) ? left : right;
#endif

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset1)) ? left : right;
#else
		next = (v > (th + offset1)) ? left : right;
#endif

		vR[i] = table[next] + offset1;

// 3nd tree
#ifdef INDIRECT
		v[0] = input[i];
#else
		v = input[i];
#endif
		next = offset2;

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset2)) ? left : right;
#else
		next = (v > (th + offset2)) ? left : right;
#endif

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset2)) ? left : right;
#else
		next = (v > (th + offset2)) ? left : right;
#endif

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset2)) ? left : right;
#else
		next = (v > (th + offset2)) ? left : right;
#endif

		vR[i] = table[next] + offset2;
	}
}
#endif

#if defined(_RF_TABLE_4TREE_GLOBAL) || defined(_RF_TABLE_4TREE_RESTRICT) || defined(_RF_TABLE_4TREE_CONST) || defined(_RF_TABLE_4TREE_SHARED)
#define _MOD 4
#define _4TREE

#if defined(_RF_TABLE_4TREE_GLOBAL)
#define EXP_NAME "_RF_TABLE_4TREE_GLOBAL"
__global__ void RF_kernel(float *input, const int n, const float *TH, const int *table, const int offset0, const int offset1, const int offset2, const int offset3, float *vR)
#elif defined(_RF_TABLE_4TREE_RESTRICT)
#define EXP_NAME "_RF_TABLE_4TREE_RESTRICT"
__global__ void RF_kernel(float *input, const int n, float *__restrict__ TH, int *__restrict__ table, const int offset0, const int offset1, const int offset2, const int offset3, float *vR)
#elif defined(_RF_TABLE_4TREE_CONST)
#define EXP_NAME "_RF_TABLE_4TREE_RESTRICT"
__global__ void RF_kernel(float *input, const int n, const int offset0, const int offset1, const int offset2, const int offset3, float *vR)
#elif defined(_RF_TABLE_4TREE_SHARED)
#define EXP_NAME "_RF_TABLE_4TREE_SHARED"
__global__ void RF_kernel(float *input, const int n, const float *t, const int *tab, const int offset0, const int offset1, const int offset2, const int offset3, float *vR)
#endif
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int next = 0, left, right;
	float th;
#ifdef _RF_TABLE_4TREE_SHARED
	__shared__ float TH[TAM_TH];
	__shared__ int table[TAM_TABLE];

	if (threadIdx.x < TAM_TH)
	{
		TH[threadIdx.x] = t[threadIdx.x];
	}
	if (threadIdx.x < TAM_TABLE)
	{
		table[threadIdx.x] = tab[threadIdx.x];
	}
	__syncthreads();
#endif

#ifdef INDIRECT
	int idx;
	int v[4];

	v[1] = 12;
	v[2] = 128;
	v[3] = 1;
#else
	int v;
#endif
	if (i < n)
	{

#ifdef INDIRECT
		v[0] = input[i];
#else
		v = input[i];
#endif

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset0)) ? left : right;
#else
		next = (v > (th + offset0)) ? left : right;
#endif

		// if (i == 2)
		//	printf("idx=%d, left=%d , right=%d , th=%f , next=%d \n", idx, left, right, th, next);

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset0)) ? left : right;
#else
		next = (v > (th + offset0)) ? left : right;
#endif

		// if (i == 2)
		//	printf("idx=%d, left=%d , right=%d , th=%f , next=%d \n", idx, left, right, th, next);

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset0)) ? left : right;
#else
		next = (v > (th + offset0)) ? left : right;
#endif

		// if (i == 2)
		//	printf("idx=%d, left=%d , right=%d , th=%f , next=%d \n", idx, left, right, th, next);

		vR[i] = table[next] + offset0;

// 2nd tree
#ifdef INDIRECT
		v[0] = input[i];
#else
		v = input[i];
#endif
		next = offset1;
#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset1)) ? left : right;
#else
		next = (v > (th + offset1)) ? left : right;
#endif

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset1)) ? left : right;
#else
		next = (v > (th + offset1)) ? left : right;
#endif

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset1)) ? left : right;
#else
		next = (v > (th + offset1)) ? left : right;
#endif

		vR[i] = table[next] + offset1;

// 3nd tree
#ifdef INDIRECT
		v[0] = input[i];
#else
		v = input[i];
#endif
		next = offset2;
#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset2)) ? left : right;
#else
		next = (v > (th + offset2)) ? left : right;
#endif

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset2)) ? left : right;
#else
		next = (v > (th + offset2)) ? left : right;
#endif

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset2)) ? left : right;
#else
		next = (v > (th + offset2)) ? left : right;
#endif

		vR[i] = table[next] + offset2;

// 4nd tree
#ifdef INDIRECT
		v[0] = input[i];
#else
		v = input[i];
#endif
		next = offset3;
#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset3)) ? left : right;
#else
		next = (v > (th + offset3)) ? left : right;
#endif

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset3)) ? left : right;
#else
		next = (v > (th + offset3)) ? left : right;
#endif

#ifdef INDIRECT
		idx = table[next] & 0x3;
#endif
		left = (table[next] >> 2) & 0xff;
		right = (table[next] >> 10) & 0xff;
		th = TH[next];
#ifdef INDIRECT
		next = (v[idx] > (th + offset3)) ? left : right;
#else
		next = (v > (th + offset3)) ? left : right;
#endif

		vR[i] = table[next] + offset3;
	}
}
#endif

void initData(float *ip, int size);
void registerTime(float value);

int main(int argc, char **argv)
{
	float elapsed_time;
	// set up device
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("[CUDA - Random Forest]: EXP %s\n", EXP_NAME);
#ifdef INDIRECT
	printf("[CUDA - Random Forest]: INDIRECT\n");
#endif
	printf("[CUDA - Random Forest]: Using Device %d: %s\n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));

	// set up data size of vectors
	int nElem = N_ELEM;
#if !defined(_RF_TABLE_1TREE_CONST) && !defined(_RF_TABLE_2TREE_CONST) && \
	!defined(_RF_TABLE_3TREE_CONST) && !defined(_RF_TABLE_4TREE_CONST)
	int tamTable = TAM_TABLE;
	int tamTh = TAM_TH;
#endif
	printf("[CUDA - Random Forest]: Vector Size %d\n", nElem);

	// malloc host memory
	size_t nBytes = nElem * sizeof(float);
#if !defined(_RF_TABLE_1TREE_CONST) && !defined(_RF_TABLE_2TREE_CONST) && \
	!defined(_RF_TABLE_3TREE_CONST) && !defined(_RF_TABLE_4TREE_CONST)
	size_t tableBytes = tamTable * sizeof(int);
	size_t thBytes = tamTh * sizeof(float);
#endif
	float *h_vR;
	h_vR = (float *)malloc(nBytes);
	float *h_input;
	h_input = (float *)malloc(nBytes);
#if !defined(_RF_TABLE_1TREE_CONST) && !defined(_RF_TABLE_2TREE_CONST) && \
	!defined(_RF_TABLE_3TREE_CONST) && !defined(_RF_TABLE_4TREE_CONST)
	int *h_table;
	h_table = (int *)malloc(tableBytes);
	float *h_th;
	h_th = (float *)malloc(thBytes);

	/*
	linha 0: th = 3, left = 1<<10, right = 2<<2, var = 0
	linha 1: th = 5, left = 3, right = 4, var = 0
	linha 2: th = 1, left = 5, right = 6, var = 0
	linha 3: th = 6, left = 7, right = 8, var = 0
	linha 4: th = 4, left = 9, right = 10, var = 0
	linha 5: th = 2, left = 11, right = 12, var = 0
	linha 6: th = 0, left = 13, right = 14, var = 0
	linha 7: 7
	linha 8: 6
	linha 9: 5
	linha 10: 4
	linha 11: 3
	linha 12: 2
	linha 13: 1
	linha 14: 0
	*/
	h_table[0] = (1 << 2) | (2 << 10);
	h_table[1] = (3 << 2) | (4 << 10);
	h_table[2] = (5 << 2) | (6 << 10);
	h_table[3] = (7 << 2) | (8 << 10);
	h_table[4] = (9 << 2) | (10 << 10);
	h_table[5] = (11 << 2) | (12 << 10);
	h_table[6] = (13 << 2) | (14 << 10);
	h_table[7] = 7;
	h_table[8] = 6;
	h_table[9] = 5;
	h_table[10] = 4;
	h_table[11] = 3;
	h_table[12] = 2;
	h_table[13] = 1;
	h_table[14] = 0;

	h_th[0] = 3;
	h_th[1] = 5;
	h_th[2] = 1;
	h_th[3] = 6;
	h_th[4] = 4;
	h_th[5] = 2;
	h_th[6] = 0;
#endif
	printf("[CUDA - Random Forest]: Start Reading Dataset\n");
	initData(h_input, nElem);
	printf("[CUDA - Random Forest]: Finished Reading Dataset\n");
	memset(h_vR, 0, nBytes);

	// malloc device global memory
	float *d_vR;
	CHECK(hipMalloc((float **)&d_vR, nBytes));
	float *d_input;
	CHECK(hipMalloc((float **)&d_input, nBytes));
#if !defined(_RF_TABLE_1TREE_CONST) && !defined(_RF_TABLE_2TREE_CONST) && \
	!defined(_RF_TABLE_3TREE_CONST) && !defined(_RF_TABLE_4TREE_CONST)
	int *d_table;
	CHECK(hipMalloc((int **)&d_table, tableBytes));
	float *d_th;
	CHECK(hipMalloc((float **)&d_th, thBytes));
#endif

	// transfer data from host to device
	CHECK(hipMemcpy(d_input, h_input, nBytes, hipMemcpyHostToDevice));
#if !defined(_RF_TABLE_1TREE_CONST) && !defined(_RF_TABLE_2TREE_CONST) && \
	!defined(_RF_TABLE_3TREE_CONST) && !defined(_RF_TABLE_4TREE_CONST)
	CHECK(hipMemcpy(d_table, h_table, tableBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_th, h_th, thBytes, hipMemcpyHostToDevice));
#else
	CHECK(cudaMemcpyToSymbol(table, tableData, TAM_TABLE * sizeof(int)));
	CHECK(cudaMemcpyToSymbol(TH, THData, TAM_TH * sizeof(float)));
#endif
	// invoke kernel at host side
	int iLen = 512; // 1024
	dim3 block(iLen);
	dim3 grid((nElem + block.x - 1) / block.x);// /2
	hipEvent_t start, stop;
	CHECK(hipEventCreate(&start));
	CHECK(hipEventCreate(&stop));

	// record start event
	CHECK(hipEventRecord(start, 0));
#if defined(_RF_TABLE_1TREE_CONST) || defined(_RF_TABLE_2TREE_CONST) || \
	defined(_RF_TABLE_3TREE_CONST) || defined(_RF_TABLE_4TREE_CONST)
#if defined(_2TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, d_vR);
#elif defined(_3TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, d_vR);
#elif defined(_4TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, d_vR);
#else
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, d_vR);
#endif
#else
#if defined(_2TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, d_th, d_table, 0, 0, d_vR);
#elif defined(_3TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, d_th, d_table, 0, 0, 0, d_vR);
#elif defined(_4TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, d_th, d_table, 0, 0, 0, 0, d_vR);
#else
	RF_kernel<<<grid, block>>>(d_input, nElem, d_th, d_table, 0, d_vR);
#endif
#endif
	CHECK(hipEventRecord(stop, 0));
	CHECK(hipEventSynchronize(stop));
	hipDeviceSynchronize();

	// calculate elapsed time
	CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
	printf("[CUDA - Random Forest]: %s - execution time = %.6fms\n", EXP_NAME, elapsed_time);
	registerTime(elapsed_time);

	// copy kernel result back to host side
	CHECK(hipMemcpy(h_vR, d_vR, nBytes, hipMemcpyDeviceToHost));
	CHECK(hipGetLastError());

	// Verify the answers
	char flag = 1;
	for (int i = 0; i < nElem; i++)
	{
		if (i < 32)
		{
			printf("%.3f ", h_vR[i]);
		}
		if (h_input[i] != h_vR[i])
		{
			flag = 0;
			// break;
		}
	}
	if (flag == 1)
	{
		printf("Return correct");
	}
	else
	{
		printf("Error!!");
	}
	printf("\n ");

	// free device memory
	hipFree(d_input);
	hipFree(d_vR);
#if !defined(_RF_TABLE_1TREE_CONST) && !defined(_RF_TABLE_2TREE_CONST) && \
	!defined(_RF_TABLE_3TREE_CONST) && !defined(_RF_TABLE_4TREE_CONST)
	hipFree(d_th);
	hipFree(d_table);
#endif

	// free host memory
	free(h_input);
	free(h_vR);
#if !defined(_RF_TABLE_1TREE_CONST) && !defined(_RF_TABLE_2TREE_CONST) && \
	!defined(_RF_TABLE_3TREE_CONST) && !defined(_RF_TABLE_4TREE_CONST)
	free(h_th);
	free(h_table);
#endif
	return (0);
}

void initData(float *ip, int size)
{
	for (int i = 0; i < size; i++)
	{
		if (WARP_SIZE < _MOD)
		{
			int aux = _MOD / WARP_SIZE;
			ip[i] = (float)((i * aux) % _MOD);
		}
		else
		{
			ip[i] = (float)(i % _MOD);
		}
		// if (i < 32)
		//	printf("%.2f ", ip[i]);
	}
	return;
}

void registerTime(float value)
{
	printf("[CUDA - Random Forest]: Time spent: %f.\n", value);
}
