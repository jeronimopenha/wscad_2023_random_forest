#include <sys/time.h>
#define CHECK(call)                                                \
	{                                                              \
		const hipError_t error = call;                            \
		if (error != hipSuccess)                                  \
		{                                                          \
			fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
			fprintf(stderr, "code: %d, reason: %s\n", error,       \
					hipGetErrorString(error));                    \
		}                                                          \
	}
#define N_ELEM 4999998
#define TH_PER_BLOCK 512
#define WARP_SIZE 32

#include <hip/hip_runtime.h>
#include <stdio.h>

void initialData(float *ip, int size)
{

	for (int i = 0; i < size; i++)
	{
		ip[i] = (float)(rand() & 15);
	}
	return;
}
FILE *inFile;
FILE *outFile;

void readInFile(float **ip)
{
	FILE *file = fopen("SUSY.csv", "r");
	if (file == NULL)
	{
		printf("[CUDA]: Failed to open the file.\n");
		exit(1);
	}

	char line[1000];

	int row = 0;
	fgets(line, sizeof(line), file);
	while (fgets(line, sizeof(line), file))
	{
		char *token = strtok(line, ",");

		int column = 0;
		while (token != NULL)
		{
			ip[column][row] = atof(token);
			token = strtok(NULL, ",");
			column++;
		}

		row++;
		if (row > N_ELEM)
			break;
	}

	fclose(file);
}
void writeOutFile(int value)
{
	outFile = fopen("out_rf.csv", "a");
	fprintf(outFile, "%d\n", value);
	fclose(outFile);
}

void registerTime(float value)
{
	outFile = fopen("results/results.csv", "a");
	fprintf(outFile, ",%.4f", value);
	fclose(outFile);
}
__global__ void RF_with_IF(float *F0, float *F1, float *F2, float *F3, float *F4, float *F5, float *F6, float *F7, float *F8,
						   float *F9, float *F10, float *F11, float *F12, float *F13, float *F14, float *F15, float *F16,
						   float *F17, float *F18, int *P, const int N, int *hist0, int *hist1, int *hist2, int *hist3)
{

	//__shared__ int th_idx[TH_PER_BLOCK];
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int Class[2];
	Class[0] = 0;
	Class[1] = 0;

	//th_idx[threadIdx.x] = 0;

	if (i < N)
	{
		int leaf;

		if (F10[i] <= (1.013))
		{
			if (F15[i] <= (0.761))
			{
				if (F7[i] <= (1.279))
				{
					if (F15[i] <= (0.575))
					{
						if (F1[i] <= (0.716))
						{
							leaf = 0;
							Class[1]++;
						}
						else
						{
							leaf = 1;
							Class[0]++;
						}
					}
					else
					{
						if (F1[i] <= (0.824))
						{
							leaf = 2;
							Class[1]++;
						}
						else
						{
							leaf = 3;
							Class[0]++;
						}
					}
				}
				else
				{
					if (F7[i] <= (1.633))
					{
						if (F13[i] <= (1.718))
						{
							leaf = 4;
							Class[0]++;
						}
						else
						{
							leaf = 5;
							Class[1]++;
						}
					}
					else
					{
						if (F10[i] <= (-0.851))
						{
							leaf = 6;
							Class[0]++;
						}
						else
						{
							leaf = 7;
							Class[0]++;
						}
					}
				}
			}
			else
			{
				if (F7[i] <= (1.156))
				{
					if (F7[i] <= (0.756))
					{
						if (F0[i] <= (0.156))
						{
							leaf = 8;
							Class[1]++;
						}
						else
						{
							leaf = 9;
							Class[0]++;
						}
					}
					else
					{
						if (F11[i] <= (0.922))
						{
							leaf = 10;
							Class[1]++;
						}
						else
						{
							leaf = 11;
							Class[0]++;
						}
					}
				}
				else
				{
					if (F1[i] <= (1.005))
					{
						if (F7[i] <= (1.57))
						{
							leaf = 12;
							Class[0]++;
						}
						else
						{
							leaf = 13;
							Class[0]++;
						}
					}
					else
					{
						if (F13[i] <= (0.699))
						{
							leaf = 14;
							Class[0]++;
						}
						else
						{
							leaf = 15;
							Class[0]++;
						}
					}
				}
			}
		}
		else
		{
			if (F7[i] <= (1.087))
			{
				if (F17[i] <= (1.312))
				{
					if (F1[i] <= (0.627))
					{
						if (F2[i] <= (-1.216))
						{
							leaf = 16;
							Class[0]++;
						}
						else
						{
							leaf = 17;
							Class[0]++;
						}
					}
					else
					{
						if (F12[i] <= (0.481))
						{
							leaf = 18;
							Class[0]++;
						}
						else
						{
							leaf = 19;
							Class[0]++;
						}
					}
				}
				else
				{
					if (F4[i] <= (1.032))
					{
						if (F11[i] <= (0.507))
						{
							leaf = 20;
							Class[1]++;
						}
						else
						{
							leaf = 21;
							Class[1]++;
						}
					}
					else
					{
						if (F17[i] <= (1.466))
						{
							leaf = 22;
							Class[0]++;
						}
						else
						{
							leaf = 23;
							Class[0]++;
						}
					}
				}
			}
			else
			{
				if (F7[i] <= (1.387))
				{
					if (F11[i] <= (0.68))
					{
						if (F17[i] <= (1.345))
						{
							leaf = 24;
							Class[0]++;
						}
						else
						{
							leaf = 25;
							Class[0]++;
						}
					}
					else
					{
						if (F2[i] <= (1.573))
						{
							leaf = 26;
							Class[0]++;
						}
						else
						{
							leaf = 27;
							Class[0]++;
						}
					}
				}
				else
				{
					if (F1[i] <= (0.723))
					{
						if (F7[i] <= (1.578))
						{
							leaf = 28;
							Class[0]++;
						}
						else
						{
							leaf = 29;
							Class[0]++;
						}
					}
					else
					{
						if (F4[i] <= (3.169))
						{
							leaf = 30;
							Class[0]++;
						}
						else
						{
							leaf = 31;
							Class[0]++;
						}
					}
				}
			}
		}

		/*th_idx[threadIdx.x] = leaf;
		if (threadIdx.x % WARP_SIZE == 0)
		{
			int uniqueValues[WARP_SIZE];
			int count = 0;

			for (int i = 0; i < WARP_SIZE; i++)
			{
				int isUnique = 1;
				for (int j = 0; j < count; j++)
				{
					if (th_idx[threadIdx.x + i] == uniqueValues[j])
					{
						isUnique = 0;
						break;
					}
				}
				if (isUnique)
				{
					uniqueValues[count] = th_idx[threadIdx.x + i];
					count++;
				}
			}*/
			//atomicAdd(&hist0[count - 1], 1);
			// if (blockIdx.x == 0 && threadIdx.x == 0)
			//	printf("%d ", hist0[count - 1]);
		//}
		

		int p0 = (Class[0] > Class[1]) ? 0 : 1;
		int Q0 = (Class[0] > Class[1]) ? Class[0] : Class[1];
		P[i] = p0;
	}
}
int main(int argc, char **argv)
{

	float elapsed_time;
	// set up device
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("[CUDA]: Using Device %d: %s\n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));

	// set up data size of vectors
	int nElem = 4999998;
	printf("[CUDA]: Vector Size %d\n", nElem);

	// malloc host memory
	size_t nBytes = nElem * sizeof(float);
	size_t histBytes = WARP_SIZE * sizeof(int);

	int *h_hist0, *h_hist1, *h_hist2, *h_hist3;
	h_hist0 = (int *)malloc(histBytes);
	h_hist1 = (int *)malloc(histBytes);
	h_hist2 = (int *)malloc(histBytes);
	h_hist3 = (int *)malloc(histBytes);

	memset(h_hist0, 0, histBytes);
	memset(h_hist1, 0, histBytes);
	memset(h_hist2, 0, histBytes);
	memset(h_hist3, 0, histBytes);

	float *h_0, *h_1, *h_2, *h_3, *h_4, *h_5, *h_6, *h_7, *h_8, *h_9, *h_10, *h_11, *h_12, *h_13, *h_14, *h_15, *h_16, *h_17, *h_18;
	int *h_P, *hostRef;
	h_0 = (float *)malloc(nBytes);
	h_1 = (float *)malloc(nBytes);
	h_2 = (float *)malloc(nBytes);
	h_3 = (float *)malloc(nBytes);
	h_4 = (float *)malloc(nBytes);
	h_5 = (float *)malloc(nBytes);
	h_6 = (float *)malloc(nBytes);
	h_7 = (float *)malloc(nBytes);
	h_8 = (float *)malloc(nBytes);
	h_9 = (float *)malloc(nBytes);
	h_10 = (float *)malloc(nBytes);
	h_11 = (float *)malloc(nBytes);
	h_12 = (float *)malloc(nBytes);
	h_13 = (float *)malloc(nBytes);
	h_14 = (float *)malloc(nBytes);
	h_15 = (float *)malloc(nBytes);
	h_16 = (float *)malloc(nBytes);
	h_17 = (float *)malloc(nBytes);
	h_18 = (float *)malloc(nBytes);
	hostRef = (int *)malloc(nBytes);
	h_P = (int *)malloc(nBytes);

	printf("[CUDA]: Start Reading Dataset\n");
	float *h[19] = {h_0, h_1, h_2, h_3, h_4, h_5, h_6, h_7, h_8, h_9, h_10, h_11, h_12, h_13, h_14, h_15, h_16, h_17, h_18};
	readInFile(h);
	printf("[CUDA]: Finished Reading Dataset\n");

	memset(hostRef, 0, nBytes);
	memset(h_P, 0, nBytes);

	// malloc device global memory
	int *d_hist0, *d_hist1, *d_hist2, *d_hist3;
	CHECK(hipMalloc((float **)&d_hist0, histBytes));
	CHECK(hipMalloc((float **)&d_hist1, histBytes));
	CHECK(hipMalloc((float **)&d_hist2, histBytes));
	CHECK(hipMalloc((float **)&d_hist3, histBytes));

	CHECK(hipMemcpy(d_hist0, h_hist0, histBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_hist1, h_hist1, histBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_hist2, h_hist2, histBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_hist3, h_hist3, histBytes, hipMemcpyHostToDevice));

	float *d_0, *d_1, *d_2, *d_3, *d_4, *d_5, *d_6, *d_7, *d_8, *d_9, *d_10, *d_11, *d_12, *d_13, *d_14, *d_15, *d_16, *d_17, *d_18;
	int *d_P;
	CHECK(hipMalloc((float **)&d_0, nBytes));
	CHECK(hipMalloc((float **)&d_1, nBytes));
	CHECK(hipMalloc((float **)&d_2, nBytes));
	CHECK(hipMalloc((float **)&d_3, nBytes));
	CHECK(hipMalloc((float **)&d_4, nBytes));
	CHECK(hipMalloc((float **)&d_5, nBytes));
	CHECK(hipMalloc((float **)&d_6, nBytes));
	CHECK(hipMalloc((float **)&d_7, nBytes));
	CHECK(hipMalloc((float **)&d_8, nBytes));
	CHECK(hipMalloc((float **)&d_9, nBytes));
	CHECK(hipMalloc((float **)&d_10, nBytes));
	CHECK(hipMalloc((float **)&d_11, nBytes));
	CHECK(hipMalloc((float **)&d_12, nBytes));
	CHECK(hipMalloc((float **)&d_13, nBytes));
	CHECK(hipMalloc((float **)&d_14, nBytes));
	CHECK(hipMalloc((float **)&d_15, nBytes));
	CHECK(hipMalloc((float **)&d_16, nBytes));
	CHECK(hipMalloc((float **)&d_17, nBytes));
	CHECK(hipMalloc((float **)&d_18, nBytes));
	CHECK(hipMalloc((int **)&d_P, nBytes));

	// transfer data from host to device
	CHECK(hipMemcpy(d_0, h_0, nBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_1, h_1, nBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_2, h_2, nBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_3, h_3, nBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_4, h_4, nBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_5, h_5, nBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_6, h_6, nBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_7, h_7, nBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_8, h_8, nBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_9, h_9, nBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_10, h_10, nBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_11, h_11, nBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_12, h_12, nBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_13, h_13, nBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_14, h_14, nBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_15, h_15, nBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_16, h_16, nBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_17, h_17, nBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_18, h_18, nBytes, hipMemcpyHostToDevice));
	// invoke kernel at host side
	int iLen = TH_PER_BLOCK;
	dim3 block(iLen);
	dim3 grid((nElem + block.x - 1) / block.x);

	hipEvent_t start, stop;
	CHECK(hipEventCreate(&start));
	CHECK(hipEventCreate(&stop));
	// record start event
	CHECK(hipEventRecord(start, 0));
	RF_with_IF<<<grid, block>>>(d_0, d_1, d_2, d_3, d_4, d_5, d_6, d_7,
								d_8, d_9, d_10, d_11, d_12, d_13, d_14,
								d_15, d_16, d_17, d_18, d_P, nElem,
								d_hist0, d_hist1, d_hist2, d_hist3);
	hipDeviceSynchronize();
	CHECK(hipEventRecord(stop, 0));
	CHECK(hipEventSynchronize(stop));
	// calculate elapsed time
	CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
	printf("[CUDA]: RF with IF - execution time = %.6fms\n", elapsed_time);

	registerTime(elapsed_time);
	CHECK(hipEventCreate(&start));
	CHECK(hipEventCreate(&stop));
	CHECK(hipGetLastError());

	// copy kernel result back to host side
	/*CHECK(cudaMemcpy(h_P, d_P, nBytes, cudaMemcpyDeviceToHost));

	CHECK(cudaMemcpy(h_hist0, d_hist0, histBytes, cudaMemcpyDeviceToHost));
	CHECK(cudaMemcpy(h_hist1, d_hist1, histBytes, cudaMemcpyDeviceToHost));
	CHECK(cudaMemcpy(h_hist2, d_hist2, histBytes, cudaMemcpyDeviceToHost));
	CHECK(cudaMemcpy(h_hist3, d_hist3, histBytes, cudaMemcpyDeviceToHost));*/

	for (int i = 0; i < WARP_SIZE; i++)
	{
		printf("%d ", h_hist0[i]);
	}
	printf("\n ");
	for (int i = 0; i < WARP_SIZE; i++)
	{
		printf("%d ", h_hist1[i]);
	}
	printf("\n ");
	for (int i = 0; i < WARP_SIZE; i++)
	{
		printf("%d ", h_hist2[i]);
	}
	printf("\n ");
	for (int i = 0; i < WARP_SIZE; i++)
	{
		printf("%d ", h_hist3[i]);
	}
	printf("\n ");

	// free host memory
	hipFree(d_0);
	hipFree(d_1);
	hipFree(d_2);
	hipFree(d_3);
	hipFree(d_4);
	hipFree(d_5);
	hipFree(d_6);
	hipFree(d_7);
	hipFree(d_8);
	hipFree(d_9);
	hipFree(d_10);
	hipFree(d_11);
	hipFree(d_12);
	hipFree(d_13);
	hipFree(d_14);
	hipFree(d_15);
	hipFree(d_16);
	hipFree(d_17);
	hipFree(d_18);
	hipFree(d_P);
	hipFree(d_hist0);
	hipFree(d_hist1);
	hipFree(d_hist2);
	hipFree(d_hist3);
	free(h_0);
	free(h_1);
	free(h_2);
	free(h_3);
	free(h_4);
	free(h_5);
	free(h_6);
	free(h_7);
	free(h_8);
	free(h_9);
	free(h_10);
	free(h_11);
	free(h_12);
	free(h_13);
	free(h_14);
	free(h_15);
	free(h_16);
	free(h_17);
	free(h_18);
	free(hostRef);
	free(h_hist0);
	free(h_hist1);
	free(h_hist2);
	free(h_hist3);

	return (0);
}