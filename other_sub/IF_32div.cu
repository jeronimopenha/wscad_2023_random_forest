#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>

#define N_ELEM 50000000
#define WARP_SIZE 32

#define CHECK(call)                                                \
    {                                                              \
        const hipError_t error = call;                            \
        if (error != hipSuccess)                                  \
        {                                                          \
            fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
            fprintf(stderr, "code: %d, reason: %s\n", error,       \
                    hipGetErrorString(error));                    \
        }                                                          \
    }

#define _MOD 32
#define _1TREE
#define EXP_NAME "_RF_IF_1TREE_5LV_TOTAL_DIVERGENCE_GLOBAL"

__global__ void
RF_kernel(float *input, const int n, const int *offset, float *vR)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (threadIdx.x%WARP_SIZE == 0)
    {
        if (input[i] > (15 + offset[0]))
        {
            if (input[i] > (23 + offset[0]))
            {
                if (input[i] > (27 + offset[0]))
                {
                    if (input[i] > (29 + offset[0]))
                    {
                        if (input[i] == (31 + offset[0]))
                        {
                            vR[i] = 31;
                        }
                        else
                        {
                            vR[i] = 30;
                        }
                    }
                    else
                    {
                        if (input[i] == (29 + offset[0]))
                        {
                            vR[i] = 29;
                        }
                        else
                        {
                            vR[i] = 28;
                        }
                    }
                }
                else
                {
                    if (input[i] > (25 + offset[0]))
                    {
                        if (input[i] == (27 + offset[0]))
                        {
                            vR[i] = 27;
                        }
                        else
                        {
                            vR[i] = 26;
                        }
                    }
                    else
                    {
                        if (input[i] == (25 + offset[0]))
                        {
                            vR[i] = 25;
                        }
                        else
                        {
                            vR[i] = 24;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (19 + offset[0]))
                {
                    if (input[i] > (21 + offset[0]))
                    {
                        if (input[i] == (23 + offset[0]))
                        {
                            vR[i] = 23;
                        }
                        else
                        {
                            vR[i] = 22;
                        }
                    }
                    else
                    {
                        if (input[i] == (21 + offset[0]))
                        {
                            vR[i] = 21;
                        }
                        else
                        {
                            vR[i] = 20;
                        }
                    }
                }
                else
                {
                    if (input[i] > (17 + offset[0]))
                    {
                        if (input[i] == (19 + offset[0]))
                        {
                            vR[i] = 19;
                        }
                        else
                        {
                            vR[i] = 18;
                        }
                    }
                    else
                    {
                        if (input[i] == (17 + offset[0]))
                        {
                            vR[i] = 17;
                        }
                        else
                        {
                            vR[i] = 16;
                        }
                    }
                }
            }
        }
        else
        {
            if (input[i] > (7 + offset[0]))
            {
                if (input[i] > (11 + offset[0]))
                {
                    if (input[i] > (13 + offset[0]))
                    {
                        if (input[i] == (15 + offset[0]))
                        {
                            vR[i] = 15;
                        }
                        else
                        {
                            vR[i] = 14;
                        }
                    }
                    else
                    {
                        if (input[i] == (13 + offset[0]))
                        {
                            vR[i] = 13;
                        }
                        else
                        {
                            vR[i] = 12;
                        }
                    }
                }
                else
                {
                    if (input[i] > (9 + offset[0]))
                    {
                        if (input[i] == (11 + offset[0]))
                        {
                            vR[i] = 11;
                        }
                        else
                        {
                            vR[i] = 10;
                        }
                    }
                    else
                    {
                        if (input[i] == (9 + offset[0]))
                        {
                            vR[i] = 9;
                        }
                        else
                        {
                            vR[i] = 8;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (3 + offset[0]))
                {
                    if (input[i] > (5 + offset[0]))
                    {
                        if (input[i] == (7 + offset[0]))
                        {
                            vR[i] = 7;
                        }
                        else
                        {
                            vR[i] = 6;
                        }
                    }
                    else
                    {
                        if (input[i] == (5 + offset[0]))
                        {
                            vR[i] = 5;
                        }
                        else
                        {
                            vR[i] = 4;
                        }
                    }
                }
                else
                {
                    if (input[i] > (1 + offset[0]))
                    {
                        if (input[i] == (3 + offset[0]))
                        {
                            vR[i] = 3;
                        }
                        else
                        {
                            vR[i] = 2;
                        }
                    }
                    else
                    {
                        if (input[i] == (1 + offset[0]))
                        {
                            vR[i] = 1;
                        }
                        else
                        {
                            vR[i] = 0;
                        }
                    }
                }
            }
        }
    }
    else if (threadIdx.x%WARP_SIZE == 1)
    {
        if (input[i] > (15 + offset[1]))
        {
            if (input[i] > (23 + offset[1]))
            {
                if (input[i] > (27 + offset[1]))
                {
                    if (input[i] > (29 + offset[1]))
                    {
                        if (input[i] == (31 + offset[1]))
                        {
                            vR[i] = 31;
                        }
                        else
                        {
                            vR[i] = 30;
                        }
                    }
                    else
                    {
                        if (input[i] == (29 + offset[1]))
                        {
                            vR[i] = 29;
                        }
                        else
                        {
                            vR[i] = 28;
                        }
                    }
                }
                else
                {
                    if (input[i] > (25 + offset[1]))
                    {
                        if (input[i] == (27 + offset[1]))
                        {
                            vR[i] = 27;
                        }
                        else
                        {
                            vR[i] = 26;
                        }
                    }
                    else
                    {
                        if (input[i] == (25 + offset[1]))
                        {
                            vR[i] = 25;
                        }
                        else
                        {
                            vR[i] = 24;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (19 + offset[1]))
                {
                    if (input[i] > (21 + offset[1]))
                    {
                        if (input[i] == (23 + offset[1]))
                        {
                            vR[i] = 23;
                        }
                        else
                        {
                            vR[i] = 22;
                        }
                    }
                    else
                    {
                        if (input[i] == (21 + offset[1]))
                        {
                            vR[i] = 21;
                        }
                        else
                        {
                            vR[i] = 20;
                        }
                    }
                }
                else
                {
                    if (input[i] > (17 + offset[1]))
                    {
                        if (input[i] == (19 + offset[1]))
                        {
                            vR[i] = 19;
                        }
                        else
                        {
                            vR[i] = 18;
                        }
                    }
                    else
                    {
                        if (input[i] == (17 + offset[1]))
                        {
                            vR[i] = 17;
                        }
                        else
                        {
                            vR[i] = 16;
                        }
                    }
                }
            }
        }
        else
        {
            if (input[i] > (7 + offset[1]))
            {
                if (input[i] > (11 + offset[1]))
                {
                    if (input[i] > (13 + offset[1]))
                    {
                        if (input[i] == (15 + offset[1]))
                        {
                            vR[i] = 15;
                        }
                        else
                        {
                            vR[i] = 14;
                        }
                    }
                    else
                    {
                        if (input[i] == (13 + offset[1]))
                        {
                            vR[i] = 13;
                        }
                        else
                        {
                            vR[i] = 12;
                        }
                    }
                }
                else
                {
                    if (input[i] > (9 + offset[1]))
                    {
                        if (input[i] == (11 + offset[1]))
                        {
                            vR[i] = 11;
                        }
                        else
                        {
                            vR[i] = 10;
                        }
                    }
                    else
                    {
                        if (input[i] == (9 + offset[1]))
                        {
                            vR[i] = 9;
                        }
                        else
                        {
                            vR[i] = 8;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (3 + offset[1]))
                {
                    if (input[i] > (5 + offset[1]))
                    {
                        if (input[i] == (7 + offset[1]))
                        {
                            vR[i] = 7;
                        }
                        else
                        {
                            vR[i] = 6;
                        }
                    }
                    else
                    {
                        if (input[i] == (5 + offset[1]))
                        {
                            vR[i] = 5;
                        }
                        else
                        {
                            vR[i] = 4;
                        }
                    }
                }
                else
                {
                    if (input[i] > (1 + offset[1]))
                    {
                        if (input[i] == (3 + offset[1]))
                        {
                            vR[i] = 3;
                        }
                        else
                        {
                            vR[i] = 2;
                        }
                    }
                    else
                    {
                        if (input[i] == (1 + offset[1]))
                        {
                            vR[i] = 1;
                        }
                        else
                        {
                            vR[i] = 0;
                        }
                    }
                }
            }
        }
    }
    else if (threadIdx.x%WARP_SIZE == 2)
    {
        if (input[i] > (15 + offset[2]))
        {
            if (input[i] > (23 + offset[2]))
            {
                if (input[i] > (27 + offset[2]))
                {
                    if (input[i] > (29 + offset[2]))
                    {
                        if (input[i] == (31 + offset[2]))
                        {
                            vR[i] = 31;
                        }
                        else
                        {
                            vR[i] = 30;
                        }
                    }
                    else
                    {
                        if (input[i] == (29 + offset[2]))
                        {
                            vR[i] = 29;
                        }
                        else
                        {
                            vR[i] = 28;
                        }
                    }
                }
                else
                {
                    if (input[i] > (25 + offset[2]))
                    {
                        if (input[i] == (27 + offset[2]))
                        {
                            vR[i] = 27;
                        }
                        else
                        {
                            vR[i] = 26;
                        }
                    }
                    else
                    {
                        if (input[i] == (25 + offset[2]))
                        {
                            vR[i] = 25;
                        }
                        else
                        {
                            vR[i] = 24;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (19 + offset[2]))
                {
                    if (input[i] > (21 + offset[2]))
                    {
                        if (input[i] == (23 + offset[2]))
                        {
                            vR[i] = 23;
                        }
                        else
                        {
                            vR[i] = 22;
                        }
                    }
                    else
                    {
                        if (input[i] == (21 + offset[2]))
                        {
                            vR[i] = 21;
                        }
                        else
                        {
                            vR[i] = 20;
                        }
                    }
                }
                else
                {
                    if (input[i] > (17 + offset[2]))
                    {
                        if (input[i] == (19 + offset[2]))
                        {
                            vR[i] = 19;
                        }
                        else
                        {
                            vR[i] = 18;
                        }
                    }
                    else
                    {
                        if (input[i] == (17 + offset[2]))
                        {
                            vR[i] = 17;
                        }
                        else
                        {
                            vR[i] = 16;
                        }
                    }
                }
            }
        }
        else
        {
            if (input[i] > (7 + offset[2]))
            {
                if (input[i] > (11 + offset[2]))
                {
                    if (input[i] > (13 + offset[2]))
                    {
                        if (input[i] == (15 + offset[2]))
                        {
                            vR[i] = 15;
                        }
                        else
                        {
                            vR[i] = 14;
                        }
                    }
                    else
                    {
                        if (input[i] == (13 + offset[2]))
                        {
                            vR[i] = 13;
                        }
                        else
                        {
                            vR[i] = 12;
                        }
                    }
                }
                else
                {
                    if (input[i] > (9 + offset[2]))
                    {
                        if (input[i] == (11 + offset[2]))
                        {
                            vR[i] = 11;
                        }
                        else
                        {
                            vR[i] = 10;
                        }
                    }
                    else
                    {
                        if (input[i] == (9 + offset[2]))
                        {
                            vR[i] = 9;
                        }
                        else
                        {
                            vR[i] = 8;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (3 + offset[2]))
                {
                    if (input[i] > (5 + offset[2]))
                    {
                        if (input[i] == (7 + offset[2]))
                        {
                            vR[i] = 7;
                        }
                        else
                        {
                            vR[i] = 6;
                        }
                    }
                    else
                    {
                        if (input[i] == (5 + offset[2]))
                        {
                            vR[i] = 5;
                        }
                        else
                        {
                            vR[i] = 4;
                        }
                    }
                }
                else
                {
                    if (input[i] > (1 + offset[2]))
                    {
                        if (input[i] == (3 + offset[2]))
                        {
                            vR[i] = 3;
                        }
                        else
                        {
                            vR[i] = 2;
                        }
                    }
                    else
                    {
                        if (input[i] == (1 + offset[2]))
                        {
                            vR[i] = 1;
                        }
                        else
                        {
                            vR[i] = 0;
                        }
                    }
                }
            }
        }
    }
    else if (threadIdx.x%WARP_SIZE == 3)
    {
        if (input[i] > (15 + offset[3]))
        {
            if (input[i] > (23 + offset[3]))
            {
                if (input[i] > (27 + offset[3]))
                {
                    if (input[i] > (29 + offset[3]))
                    {
                        if (input[i] == (31 + offset[3]))
                        {
                            vR[i] = 31;
                        }
                        else
                        {
                            vR[i] = 30;
                        }
                    }
                    else
                    {
                        if (input[i] == (29 + offset[3]))
                        {
                            vR[i] = 29;
                        }
                        else
                        {
                            vR[i] = 28;
                        }
                    }
                }
                else
                {
                    if (input[i] > (25 + offset[3]))
                    {
                        if (input[i] == (27 + offset[3]))
                        {
                            vR[i] = 27;
                        }
                        else
                        {
                            vR[i] = 26;
                        }
                    }
                    else
                    {
                        if (input[i] == (25 + offset[3]))
                        {
                            vR[i] = 25;
                        }
                        else
                        {
                            vR[i] = 24;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (19 + offset[3]))
                {
                    if (input[i] > (21 + offset[3]))
                    {
                        if (input[i] == (23 + offset[3]))
                        {
                            vR[i] = 23;
                        }
                        else
                        {
                            vR[i] = 22;
                        }
                    }
                    else
                    {
                        if (input[i] == (21 + offset[3]))
                        {
                            vR[i] = 21;
                        }
                        else
                        {
                            vR[i] = 20;
                        }
                    }
                }
                else
                {
                    if (input[i] > (17 + offset[3]))
                    {
                        if (input[i] == (19 + offset[3]))
                        {
                            vR[i] = 19;
                        }
                        else
                        {
                            vR[i] = 18;
                        }
                    }
                    else
                    {
                        if (input[i] == (17 + offset[3]))
                        {
                            vR[i] = 17;
                        }
                        else
                        {
                            vR[i] = 16;
                        }
                    }
                }
            }
        }
        else
        {
            if (input[i] > (7 + offset[3]))
            {
                if (input[i] > (11 + offset[3]))
                {
                    if (input[i] > (13 + offset[3]))
                    {
                        if (input[i] == (15 + offset[3]))
                        {
                            vR[i] = 15;
                        }
                        else
                        {
                            vR[i] = 14;
                        }
                    }
                    else
                    {
                        if (input[i] == (13 + offset[3]))
                        {
                            vR[i] = 13;
                        }
                        else
                        {
                            vR[i] = 12;
                        }
                    }
                }
                else
                {
                    if (input[i] > (9 + offset[3]))
                    {
                        if (input[i] == (11 + offset[3]))
                        {
                            vR[i] = 11;
                        }
                        else
                        {
                            vR[i] = 10;
                        }
                    }
                    else
                    {
                        if (input[i] == (9 + offset[3]))
                        {
                            vR[i] = 9;
                        }
                        else
                        {
                            vR[i] = 8;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (3 + offset[3]))
                {
                    if (input[i] > (5 + offset[3]))
                    {
                        if (input[i] == (7 + offset[3]))
                        {
                            vR[i] = 7;
                        }
                        else
                        {
                            vR[i] = 6;
                        }
                    }
                    else
                    {
                        if (input[i] == (5 + offset[3]))
                        {
                            vR[i] = 5;
                        }
                        else
                        {
                            vR[i] = 4;
                        }
                    }
                }
                else
                {
                    if (input[i] > (1 + offset[3]))
                    {
                        if (input[i] == (3 + offset[3]))
                        {
                            vR[i] = 3;
                        }
                        else
                        {
                            vR[i] = 2;
                        }
                    }
                    else
                    {
                        if (input[i] == (1 + offset[3]))
                        {
                            vR[i] = 1;
                        }
                        else
                        {
                            vR[i] = 0;
                        }
                    }
                }
            }
        }
    }
    else if (threadIdx.x%WARP_SIZE == 4)
    {
        if (input[i] > (15 + offset[4]))
        {
            if (input[i] > (23 + offset[4]))
            {
                if (input[i] > (27 + offset[4]))
                {
                    if (input[i] > (29 + offset[4]))
                    {
                        if (input[i] == (31 + offset[4]))
                        {
                            vR[i] = 31;
                        }
                        else
                        {
                            vR[i] = 30;
                        }
                    }
                    else
                    {
                        if (input[i] == (29 + offset[4]))
                        {
                            vR[i] = 29;
                        }
                        else
                        {
                            vR[i] = 28;
                        }
                    }
                }
                else
                {
                    if (input[i] > (25 + offset[4]))
                    {
                        if (input[i] == (27 + offset[4]))
                        {
                            vR[i] = 27;
                        }
                        else
                        {
                            vR[i] = 26;
                        }
                    }
                    else
                    {
                        if (input[i] == (25 + offset[4]))
                        {
                            vR[i] = 25;
                        }
                        else
                        {
                            vR[i] = 24;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (19 + offset[4]))
                {
                    if (input[i] > (21 + offset[4]))
                    {
                        if (input[i] == (23 + offset[4]))
                        {
                            vR[i] = 23;
                        }
                        else
                        {
                            vR[i] = 22;
                        }
                    }
                    else
                    {
                        if (input[i] == (21 + offset[4]))
                        {
                            vR[i] = 21;
                        }
                        else
                        {
                            vR[i] = 20;
                        }
                    }
                }
                else
                {
                    if (input[i] > (17 + offset[4]))
                    {
                        if (input[i] == (19 + offset[4]))
                        {
                            vR[i] = 19;
                        }
                        else
                        {
                            vR[i] = 18;
                        }
                    }
                    else
                    {
                        if (input[i] == (17 + offset[4]))
                        {
                            vR[i] = 17;
                        }
                        else
                        {
                            vR[i] = 16;
                        }
                    }
                }
            }
        }
        else
        {
            if (input[i] > (7 + offset[4]))
            {
                if (input[i] > (11 + offset[4]))
                {
                    if (input[i] > (13 + offset[4]))
                    {
                        if (input[i] == (15 + offset[4]))
                        {
                            vR[i] = 15;
                        }
                        else
                        {
                            vR[i] = 14;
                        }
                    }
                    else
                    {
                        if (input[i] == (13 + offset[4]))
                        {
                            vR[i] = 13;
                        }
                        else
                        {
                            vR[i] = 12;
                        }
                    }
                }
                else
                {
                    if (input[i] > (9 + offset[4]))
                    {
                        if (input[i] == (11 + offset[4]))
                        {
                            vR[i] = 11;
                        }
                        else
                        {
                            vR[i] = 10;
                        }
                    }
                    else
                    {
                        if (input[i] == (9 + offset[4]))
                        {
                            vR[i] = 9;
                        }
                        else
                        {
                            vR[i] = 8;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (3 + offset[4]))
                {
                    if (input[i] > (5 + offset[4]))
                    {
                        if (input[i] == (7 + offset[4]))
                        {
                            vR[i] = 7;
                        }
                        else
                        {
                            vR[i] = 6;
                        }
                    }
                    else
                    {
                        if (input[i] == (5 + offset[4]))
                        {
                            vR[i] = 5;
                        }
                        else
                        {
                            vR[i] = 4;
                        }
                    }
                }
                else
                {
                    if (input[i] > (1 + offset[4]))
                    {
                        if (input[i] == (3 + offset[4]))
                        {
                            vR[i] = 3;
                        }
                        else
                        {
                            vR[i] = 2;
                        }
                    }
                    else
                    {
                        if (input[i] == (1 + offset[4]))
                        {
                            vR[i] = 1;
                        }
                        else
                        {
                            vR[i] = 0;
                        }
                    }
                }
            }
        }
    }
    else if (threadIdx.x%WARP_SIZE == 5)
    {
        if (input[i] > (15 + offset[5]))
        {
            if (input[i] > (23 + offset[5]))
            {
                if (input[i] > (27 + offset[5]))
                {
                    if (input[i] > (29 + offset[5]))
                    {
                        if (input[i] == (31 + offset[5]))
                        {
                            vR[i] = 31;
                        }
                        else
                        {
                            vR[i] = 30;
                        }
                    }
                    else
                    {
                        if (input[i] == (29 + offset[5]))
                        {
                            vR[i] = 29;
                        }
                        else
                        {
                            vR[i] = 28;
                        }
                    }
                }
                else
                {
                    if (input[i] > (25 + offset[5]))
                    {
                        if (input[i] == (27 + offset[5]))
                        {
                            vR[i] = 27;
                        }
                        else
                        {
                            vR[i] = 26;
                        }
                    }
                    else
                    {
                        if (input[i] == (25 + offset[5]))
                        {
                            vR[i] = 25;
                        }
                        else
                        {
                            vR[i] = 24;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (19 + offset[5]))
                {
                    if (input[i] > (21 + offset[5]))
                    {
                        if (input[i] == (23 + offset[5]))
                        {
                            vR[i] = 23;
                        }
                        else
                        {
                            vR[i] = 22;
                        }
                    }
                    else
                    {
                        if (input[i] == (21 + offset[5]))
                        {
                            vR[i] = 21;
                        }
                        else
                        {
                            vR[i] = 20;
                        }
                    }
                }
                else
                {
                    if (input[i] > (17 + offset[5]))
                    {
                        if (input[i] == (19 + offset[5]))
                        {
                            vR[i] = 19;
                        }
                        else
                        {
                            vR[i] = 18;
                        }
                    }
                    else
                    {
                        if (input[i] == (17 + offset[5]))
                        {
                            vR[i] = 17;
                        }
                        else
                        {
                            vR[i] = 16;
                        }
                    }
                }
            }
        }
        else
        {
            if (input[i] > (7 + offset[5]))
            {
                if (input[i] > (11 + offset[5]))
                {
                    if (input[i] > (13 + offset[5]))
                    {
                        if (input[i] == (15 + offset[5]))
                        {
                            vR[i] = 15;
                        }
                        else
                        {
                            vR[i] = 14;
                        }
                    }
                    else
                    {
                        if (input[i] == (13 + offset[5]))
                        {
                            vR[i] = 13;
                        }
                        else
                        {
                            vR[i] = 12;
                        }
                    }
                }
                else
                {
                    if (input[i] > (9 + offset[5]))
                    {
                        if (input[i] == (11 + offset[5]))
                        {
                            vR[i] = 11;
                        }
                        else
                        {
                            vR[i] = 10;
                        }
                    }
                    else
                    {
                        if (input[i] == (9 + offset[5]))
                        {
                            vR[i] = 9;
                        }
                        else
                        {
                            vR[i] = 8;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (3 + offset[5]))
                {
                    if (input[i] > (5 + offset[5]))
                    {
                        if (input[i] == (7 + offset[5]))
                        {
                            vR[i] = 7;
                        }
                        else
                        {
                            vR[i] = 6;
                        }
                    }
                    else
                    {
                        if (input[i] == (5 + offset[5]))
                        {
                            vR[i] = 5;
                        }
                        else
                        {
                            vR[i] = 4;
                        }
                    }
                }
                else
                {
                    if (input[i] > (1 + offset[5]))
                    {
                        if (input[i] == (3 + offset[5]))
                        {
                            vR[i] = 3;
                        }
                        else
                        {
                            vR[i] = 2;
                        }
                    }
                    else
                    {
                        if (input[i] == (1 + offset[5]))
                        {
                            vR[i] = 1;
                        }
                        else
                        {
                            vR[i] = 0;
                        }
                    }
                }
            }
        }
    }
    else if (threadIdx.x%WARP_SIZE == 6)
    {
        if (input[i] > (15 + offset[6]))
        {
            if (input[i] > (23 + offset[6]))
            {
                if (input[i] > (27 + offset[6]))
                {
                    if (input[i] > (29 + offset[6]))
                    {
                        if (input[i] == (31 + offset[6]))
                        {
                            vR[i] = 31;
                        }
                        else
                        {
                            vR[i] = 30;
                        }
                    }
                    else
                    {
                        if (input[i] == (29 + offset[6]))
                        {
                            vR[i] = 29;
                        }
                        else
                        {
                            vR[i] = 28;
                        }
                    }
                }
                else
                {
                    if (input[i] > (25 + offset[6]))
                    {
                        if (input[i] == (27 + offset[6]))
                        {
                            vR[i] = 27;
                        }
                        else
                        {
                            vR[i] = 26;
                        }
                    }
                    else
                    {
                        if (input[i] == (25 + offset[6]))
                        {
                            vR[i] = 25;
                        }
                        else
                        {
                            vR[i] = 24;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (19 + offset[6]))
                {
                    if (input[i] > (21 + offset[6]))
                    {
                        if (input[i] == (23 + offset[6]))
                        {
                            vR[i] = 23;
                        }
                        else
                        {
                            vR[i] = 22;
                        }
                    }
                    else
                    {
                        if (input[i] == (21 + offset[6]))
                        {
                            vR[i] = 21;
                        }
                        else
                        {
                            vR[i] = 20;
                        }
                    }
                }
                else
                {
                    if (input[i] > (17 + offset[6]))
                    {
                        if (input[i] == (19 + offset[6]))
                        {
                            vR[i] = 19;
                        }
                        else
                        {
                            vR[i] = 18;
                        }
                    }
                    else
                    {
                        if (input[i] == (17 + offset[6]))
                        {
                            vR[i] = 17;
                        }
                        else
                        {
                            vR[i] = 16;
                        }
                    }
                }
            }
        }
        else
        {
            if (input[i] > (7 + offset[6]))
            {
                if (input[i] > (11 + offset[6]))
                {
                    if (input[i] > (13 + offset[6]))
                    {
                        if (input[i] == (15 + offset[6]))
                        {
                            vR[i] = 15;
                        }
                        else
                        {
                            vR[i] = 14;
                        }
                    }
                    else
                    {
                        if (input[i] == (13 + offset[6]))
                        {
                            vR[i] = 13;
                        }
                        else
                        {
                            vR[i] = 12;
                        }
                    }
                }
                else
                {
                    if (input[i] > (9 + offset[6]))
                    {
                        if (input[i] == (11 + offset[6]))
                        {
                            vR[i] = 11;
                        }
                        else
                        {
                            vR[i] = 10;
                        }
                    }
                    else
                    {
                        if (input[i] == (9 + offset[6]))
                        {
                            vR[i] = 9;
                        }
                        else
                        {
                            vR[i] = 8;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (3 + offset[6]))
                {
                    if (input[i] > (5 + offset[6]))
                    {
                        if (input[i] == (7 + offset[6]))
                        {
                            vR[i] = 7;
                        }
                        else
                        {
                            vR[i] = 6;
                        }
                    }
                    else
                    {
                        if (input[i] == (5 + offset[6]))
                        {
                            vR[i] = 5;
                        }
                        else
                        {
                            vR[i] = 4;
                        }
                    }
                }
                else
                {
                    if (input[i] > (1 + offset[6]))
                    {
                        if (input[i] == (3 + offset[6]))
                        {
                            vR[i] = 3;
                        }
                        else
                        {
                            vR[i] = 2;
                        }
                    }
                    else
                    {
                        if (input[i] == (1 + offset[6]))
                        {
                            vR[i] = 1;
                        }
                        else
                        {
                            vR[i] = 0;
                        }
                    }
                }
            }
        }
    }
    else if (threadIdx.x%WARP_SIZE == 7)
    {
        if (input[i] > (15 + offset[7]))
        {
            if (input[i] > (23 + offset[7]))
            {
                if (input[i] > (27 + offset[7]))
                {
                    if (input[i] > (29 + offset[7]))
                    {
                        if (input[i] == (31 + offset[7]))
                        {
                            vR[i] = 31;
                        }
                        else
                        {
                            vR[i] = 30;
                        }
                    }
                    else
                    {
                        if (input[i] == (29 + offset[7]))
                        {
                            vR[i] = 29;
                        }
                        else
                        {
                            vR[i] = 28;
                        }
                    }
                }
                else
                {
                    if (input[i] > (25 + offset[7]))
                    {
                        if (input[i] == (27 + offset[7]))
                        {
                            vR[i] = 27;
                        }
                        else
                        {
                            vR[i] = 26;
                        }
                    }
                    else
                    {
                        if (input[i] == (25 + offset[7]))
                        {
                            vR[i] = 25;
                        }
                        else
                        {
                            vR[i] = 24;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (19 + offset[7]))
                {
                    if (input[i] > (21 + offset[7]))
                    {
                        if (input[i] == (23 + offset[7]))
                        {
                            vR[i] = 23;
                        }
                        else
                        {
                            vR[i] = 22;
                        }
                    }
                    else
                    {
                        if (input[i] == (21 + offset[7]))
                        {
                            vR[i] = 21;
                        }
                        else
                        {
                            vR[i] = 20;
                        }
                    }
                }
                else
                {
                    if (input[i] > (17 + offset[7]))
                    {
                        if (input[i] == (19 + offset[7]))
                        {
                            vR[i] = 19;
                        }
                        else
                        {
                            vR[i] = 18;
                        }
                    }
                    else
                    {
                        if (input[i] == (17 + offset[7]))
                        {
                            vR[i] = 17;
                        }
                        else
                        {
                            vR[i] = 16;
                        }
                    }
                }
            }
        }
        else
        {
            if (input[i] > (7 + offset[7]))
            {
                if (input[i] > (11 + offset[7]))
                {
                    if (input[i] > (13 + offset[7]))
                    {
                        if (input[i] == (15 + offset[7]))
                        {
                            vR[i] = 15;
                        }
                        else
                        {
                            vR[i] = 14;
                        }
                    }
                    else
                    {
                        if (input[i] == (13 + offset[7]))
                        {
                            vR[i] = 13;
                        }
                        else
                        {
                            vR[i] = 12;
                        }
                    }
                }
                else
                {
                    if (input[i] > (9 + offset[7]))
                    {
                        if (input[i] == (11 + offset[7]))
                        {
                            vR[i] = 11;
                        }
                        else
                        {
                            vR[i] = 10;
                        }
                    }
                    else
                    {
                        if (input[i] == (9 + offset[7]))
                        {
                            vR[i] = 9;
                        }
                        else
                        {
                            vR[i] = 8;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (3 + offset[7]))
                {
                    if (input[i] > (5 + offset[7]))
                    {
                        if (input[i] == (7 + offset[7]))
                        {
                            vR[i] = 7;
                        }
                        else
                        {
                            vR[i] = 6;
                        }
                    }
                    else
                    {
                        if (input[i] == (5 + offset[7]))
                        {
                            vR[i] = 5;
                        }
                        else
                        {
                            vR[i] = 4;
                        }
                    }
                }
                else
                {
                    if (input[i] > (1 + offset[7]))
                    {
                        if (input[i] == (3 + offset[7]))
                        {
                            vR[i] = 3;
                        }
                        else
                        {
                            vR[i] = 2;
                        }
                    }
                    else
                    {
                        if (input[i] == (1 + offset[7]))
                        {
                            vR[i] = 1;
                        }
                        else
                        {
                            vR[i] = 0;
                        }
                    }
                }
            }
        }
    }
    else if (threadIdx.x%WARP_SIZE == 8)
    {
        if (input[i] > (15 + offset[8]))
        {
            if (input[i] > (23 + offset[8]))
            {
                if (input[i] > (27 + offset[8]))
                {
                    if (input[i] > (29 + offset[8]))
                    {
                        if (input[i] == (31 + offset[8]))
                        {
                            vR[i] = 31;
                        }
                        else
                        {
                            vR[i] = 30;
                        }
                    }
                    else
                    {
                        if (input[i] == (29 + offset[8]))
                        {
                            vR[i] = 29;
                        }
                        else
                        {
                            vR[i] = 28;
                        }
                    }
                }
                else
                {
                    if (input[i] > (25 + offset[8]))
                    {
                        if (input[i] == (27 + offset[8]))
                        {
                            vR[i] = 27;
                        }
                        else
                        {
                            vR[i] = 26;
                        }
                    }
                    else
                    {
                        if (input[i] == (25 + offset[8]))
                        {
                            vR[i] = 25;
                        }
                        else
                        {
                            vR[i] = 24;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (19 + offset[8]))
                {
                    if (input[i] > (21 + offset[8]))
                    {
                        if (input[i] == (23 + offset[8]))
                        {
                            vR[i] = 23;
                        }
                        else
                        {
                            vR[i] = 22;
                        }
                    }
                    else
                    {
                        if (input[i] == (21 + offset[8]))
                        {
                            vR[i] = 21;
                        }
                        else
                        {
                            vR[i] = 20;
                        }
                    }
                }
                else
                {
                    if (input[i] > (17 + offset[8]))
                    {
                        if (input[i] == (19 + offset[8]))
                        {
                            vR[i] = 19;
                        }
                        else
                        {
                            vR[i] = 18;
                        }
                    }
                    else
                    {
                        if (input[i] == (17 + offset[8]))
                        {
                            vR[i] = 17;
                        }
                        else
                        {
                            vR[i] = 16;
                        }
                    }
                }
            }
        }
        else
        {
            if (input[i] > (7 + offset[8]))
            {
                if (input[i] > (11 + offset[8]))
                {
                    if (input[i] > (13 + offset[8]))
                    {
                        if (input[i] == (15 + offset[8]))
                        {
                            vR[i] = 15;
                        }
                        else
                        {
                            vR[i] = 14;
                        }
                    }
                    else
                    {
                        if (input[i] == (13 + offset[8]))
                        {
                            vR[i] = 13;
                        }
                        else
                        {
                            vR[i] = 12;
                        }
                    }
                }
                else
                {
                    if (input[i] > (9 + offset[8]))
                    {
                        if (input[i] == (11 + offset[8]))
                        {
                            vR[i] = 11;
                        }
                        else
                        {
                            vR[i] = 10;
                        }
                    }
                    else
                    {
                        if (input[i] == (9 + offset[8]))
                        {
                            vR[i] = 9;
                        }
                        else
                        {
                            vR[i] = 8;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (3 + offset[8]))
                {
                    if (input[i] > (5 + offset[8]))
                    {
                        if (input[i] == (7 + offset[8]))
                        {
                            vR[i] = 7;
                        }
                        else
                        {
                            vR[i] = 6;
                        }
                    }
                    else
                    {
                        if (input[i] == (5 + offset[8]))
                        {
                            vR[i] = 5;
                        }
                        else
                        {
                            vR[i] = 4;
                        }
                    }
                }
                else
                {
                    if (input[i] > (1 + offset[8]))
                    {
                        if (input[i] == (3 + offset[8]))
                        {
                            vR[i] = 3;
                        }
                        else
                        {
                            vR[i] = 2;
                        }
                    }
                    else
                    {
                        if (input[i] == (1 + offset[8]))
                        {
                            vR[i] = 1;
                        }
                        else
                        {
                            vR[i] = 0;
                        }
                    }
                }
            }
        }
    }
    else if (threadIdx.x%WARP_SIZE == 9)
    {
        if (input[i] > (15 + offset[9]))
        {
            if (input[i] > (23 + offset[9]))
            {
                if (input[i] > (27 + offset[9]))
                {
                    if (input[i] > (29 + offset[9]))
                    {
                        if (input[i] == (31 + offset[9]))
                        {
                            vR[i] = 31;
                        }
                        else
                        {
                            vR[i] = 30;
                        }
                    }
                    else
                    {
                        if (input[i] == (29 + offset[9]))
                        {
                            vR[i] = 29;
                        }
                        else
                        {
                            vR[i] = 28;
                        }
                    }
                }
                else
                {
                    if (input[i] > (25 + offset[9]))
                    {
                        if (input[i] == (27 + offset[9]))
                        {
                            vR[i] = 27;
                        }
                        else
                        {
                            vR[i] = 26;
                        }
                    }
                    else
                    {
                        if (input[i] == (25 + offset[9]))
                        {
                            vR[i] = 25;
                        }
                        else
                        {
                            vR[i] = 24;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (19 + offset[9]))
                {
                    if (input[i] > (21 + offset[9]))
                    {
                        if (input[i] == (23 + offset[9]))
                        {
                            vR[i] = 23;
                        }
                        else
                        {
                            vR[i] = 22;
                        }
                    }
                    else
                    {
                        if (input[i] == (21 + offset[9]))
                        {
                            vR[i] = 21;
                        }
                        else
                        {
                            vR[i] = 20;
                        }
                    }
                }
                else
                {
                    if (input[i] > (17 + offset[9]))
                    {
                        if (input[i] == (19 + offset[9]))
                        {
                            vR[i] = 19;
                        }
                        else
                        {
                            vR[i] = 18;
                        }
                    }
                    else
                    {
                        if (input[i] == (17 + offset[9]))
                        {
                            vR[i] = 17;
                        }
                        else
                        {
                            vR[i] = 16;
                        }
                    }
                }
            }
        }
        else
        {
            if (input[i] > (7 + offset[9]))
            {
                if (input[i] > (11 + offset[9]))
                {
                    if (input[i] > (13 + offset[9]))
                    {
                        if (input[i] == (15 + offset[9]))
                        {
                            vR[i] = 15;
                        }
                        else
                        {
                            vR[i] = 14;
                        }
                    }
                    else
                    {
                        if (input[i] == (13 + offset[9]))
                        {
                            vR[i] = 13;
                        }
                        else
                        {
                            vR[i] = 12;
                        }
                    }
                }
                else
                {
                    if (input[i] > (9 + offset[9]))
                    {
                        if (input[i] == (11 + offset[9]))
                        {
                            vR[i] = 11;
                        }
                        else
                        {
                            vR[i] = 10;
                        }
                    }
                    else
                    {
                        if (input[i] == (9 + offset[9]))
                        {
                            vR[i] = 9;
                        }
                        else
                        {
                            vR[i] = 8;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (3 + offset[9]))
                {
                    if (input[i] > (5 + offset[9]))
                    {
                        if (input[i] == (7 + offset[9]))
                        {
                            vR[i] = 7;
                        }
                        else
                        {
                            vR[i] = 6;
                        }
                    }
                    else
                    {
                        if (input[i] == (5 + offset[9]))
                        {
                            vR[i] = 5;
                        }
                        else
                        {
                            vR[i] = 4;
                        }
                    }
                }
                else
                {
                    if (input[i] > (1 + offset[9]))
                    {
                        if (input[i] == (3 + offset[9]))
                        {
                            vR[i] = 3;
                        }
                        else
                        {
                            vR[i] = 2;
                        }
                    }
                    else
                    {
                        if (input[i] == (1 + offset[9]))
                        {
                            vR[i] = 1;
                        }
                        else
                        {
                            vR[i] = 0;
                        }
                    }
                }
            }
        }
    }
    else if (threadIdx.x%WARP_SIZE == 10)
    {
        if (input[i] > (15 + offset[10]))
        {
            if (input[i] > (23 + offset[10]))
            {
                if (input[i] > (27 + offset[10]))
                {
                    if (input[i] > (29 + offset[10]))
                    {
                        if (input[i] == (31 + offset[10]))
                        {
                            vR[i] = 31;
                        }
                        else
                        {
                            vR[i] = 30;
                        }
                    }
                    else
                    {
                        if (input[i] == (29 + offset[10]))
                        {
                            vR[i] = 29;
                        }
                        else
                        {
                            vR[i] = 28;
                        }
                    }
                }
                else
                {
                    if (input[i] > (25 + offset[10]))
                    {
                        if (input[i] == (27 + offset[10]))
                        {
                            vR[i] = 27;
                        }
                        else
                        {
                            vR[i] = 26;
                        }
                    }
                    else
                    {
                        if (input[i] == (25 + offset[10]))
                        {
                            vR[i] = 25;
                        }
                        else
                        {
                            vR[i] = 24;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (19 + offset[10]))
                {
                    if (input[i] > (21 + offset[10]))
                    {
                        if (input[i] == (23 + offset[10]))
                        {
                            vR[i] = 23;
                        }
                        else
                        {
                            vR[i] = 22;
                        }
                    }
                    else
                    {
                        if (input[i] == (21 + offset[10]))
                        {
                            vR[i] = 21;
                        }
                        else
                        {
                            vR[i] = 20;
                        }
                    }
                }
                else
                {
                    if (input[i] > (17 + offset[10]))
                    {
                        if (input[i] == (19 + offset[10]))
                        {
                            vR[i] = 19;
                        }
                        else
                        {
                            vR[i] = 18;
                        }
                    }
                    else
                    {
                        if (input[i] == (17 + offset[10]))
                        {
                            vR[i] = 17;
                        }
                        else
                        {
                            vR[i] = 16;
                        }
                    }
                }
            }
        }
        else
        {
            if (input[i] > (7 + offset[10]))
            {
                if (input[i] > (11 + offset[10]))
                {
                    if (input[i] > (13 + offset[10]))
                    {
                        if (input[i] == (15 + offset[10]))
                        {
                            vR[i] = 15;
                        }
                        else
                        {
                            vR[i] = 14;
                        }
                    }
                    else
                    {
                        if (input[i] == (13 + offset[10]))
                        {
                            vR[i] = 13;
                        }
                        else
                        {
                            vR[i] = 12;
                        }
                    }
                }
                else
                {
                    if (input[i] > (9 + offset[10]))
                    {
                        if (input[i] == (11 + offset[10]))
                        {
                            vR[i] = 11;
                        }
                        else
                        {
                            vR[i] = 10;
                        }
                    }
                    else
                    {
                        if (input[i] == (9 + offset[10]))
                        {
                            vR[i] = 9;
                        }
                        else
                        {
                            vR[i] = 8;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (3 + offset[10]))
                {
                    if (input[i] > (5 + offset[10]))
                    {
                        if (input[i] == (7 + offset[10]))
                        {
                            vR[i] = 7;
                        }
                        else
                        {
                            vR[i] = 6;
                        }
                    }
                    else
                    {
                        if (input[i] == (5 + offset[10]))
                        {
                            vR[i] = 5;
                        }
                        else
                        {
                            vR[i] = 4;
                        }
                    }
                }
                else
                {
                    if (input[i] > (1 + offset[10]))
                    {
                        if (input[i] == (3 + offset[10]))
                        {
                            vR[i] = 3;
                        }
                        else
                        {
                            vR[i] = 2;
                        }
                    }
                    else
                    {
                        if (input[i] == (1 + offset[10]))
                        {
                            vR[i] = 1;
                        }
                        else
                        {
                            vR[i] = 0;
                        }
                    }
                }
            }
        }
    }
    else if (threadIdx.x%WARP_SIZE == 11)
    {
        if (input[i] > (15 + offset[11]))
        {
            if (input[i] > (23 + offset[11]))
            {
                if (input[i] > (27 + offset[11]))
                {
                    if (input[i] > (29 + offset[11]))
                    {
                        if (input[i] == (31 + offset[11]))
                        {
                            vR[i] = 31;
                        }
                        else
                        {
                            vR[i] = 30;
                        }
                    }
                    else
                    {
                        if (input[i] == (29 + offset[11]))
                        {
                            vR[i] = 29;
                        }
                        else
                        {
                            vR[i] = 28;
                        }
                    }
                }
                else
                {
                    if (input[i] > (25 + offset[11]))
                    {
                        if (input[i] == (27 + offset[11]))
                        {
                            vR[i] = 27;
                        }
                        else
                        {
                            vR[i] = 26;
                        }
                    }
                    else
                    {
                        if (input[i] == (25 + offset[11]))
                        {
                            vR[i] = 25;
                        }
                        else
                        {
                            vR[i] = 24;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (19 + offset[11]))
                {
                    if (input[i] > (21 + offset[11]))
                    {
                        if (input[i] == (23 + offset[11]))
                        {
                            vR[i] = 23;
                        }
                        else
                        {
                            vR[i] = 22;
                        }
                    }
                    else
                    {
                        if (input[i] == (21 + offset[11]))
                        {
                            vR[i] = 21;
                        }
                        else
                        {
                            vR[i] = 20;
                        }
                    }
                }
                else
                {
                    if (input[i] > (17 + offset[11]))
                    {
                        if (input[i] == (19 + offset[11]))
                        {
                            vR[i] = 19;
                        }
                        else
                        {
                            vR[i] = 18;
                        }
                    }
                    else
                    {
                        if (input[i] == (17 + offset[11]))
                        {
                            vR[i] = 17;
                        }
                        else
                        {
                            vR[i] = 16;
                        }
                    }
                }
            }
        }
        else
        {
            if (input[i] > (7 + offset[11]))
            {
                if (input[i] > (11 + offset[11]))
                {
                    if (input[i] > (13 + offset[11]))
                    {
                        if (input[i] == (15 + offset[11]))
                        {
                            vR[i] = 15;
                        }
                        else
                        {
                            vR[i] = 14;
                        }
                    }
                    else
                    {
                        if (input[i] == (13 + offset[11]))
                        {
                            vR[i] = 13;
                        }
                        else
                        {
                            vR[i] = 12;
                        }
                    }
                }
                else
                {
                    if (input[i] > (9 + offset[11]))
                    {
                        if (input[i] == (11 + offset[11]))
                        {
                            vR[i] = 11;
                        }
                        else
                        {
                            vR[i] = 10;
                        }
                    }
                    else
                    {
                        if (input[i] == (9 + offset[11]))
                        {
                            vR[i] = 9;
                        }
                        else
                        {
                            vR[i] = 8;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (3 + offset[11]))
                {
                    if (input[i] > (5 + offset[11]))
                    {
                        if (input[i] == (7 + offset[11]))
                        {
                            vR[i] = 7;
                        }
                        else
                        {
                            vR[i] = 6;
                        }
                    }
                    else
                    {
                        if (input[i] == (5 + offset[11]))
                        {
                            vR[i] = 5;
                        }
                        else
                        {
                            vR[i] = 4;
                        }
                    }
                }
                else
                {
                    if (input[i] > (1 + offset[11]))
                    {
                        if (input[i] == (3 + offset[11]))
                        {
                            vR[i] = 3;
                        }
                        else
                        {
                            vR[i] = 2;
                        }
                    }
                    else
                    {
                        if (input[i] == (1 + offset[11]))
                        {
                            vR[i] = 1;
                        }
                        else
                        {
                            vR[i] = 0;
                        }
                    }
                }
            }
        }
    }
    else if (threadIdx.x%WARP_SIZE == 12)
    {
        if (input[i] > (15 + offset[12]))
        {
            if (input[i] > (23 + offset[12]))
            {
                if (input[i] > (27 + offset[12]))
                {
                    if (input[i] > (29 + offset[12]))
                    {
                        if (input[i] == (31 + offset[12]))
                        {
                            vR[i] = 31;
                        }
                        else
                        {
                            vR[i] = 30;
                        }
                    }
                    else
                    {
                        if (input[i] == (29 + offset[12]))
                        {
                            vR[i] = 29;
                        }
                        else
                        {
                            vR[i] = 28;
                        }
                    }
                }
                else
                {
                    if (input[i] > (25 + offset[12]))
                    {
                        if (input[i] == (27 + offset[12]))
                        {
                            vR[i] = 27;
                        }
                        else
                        {
                            vR[i] = 26;
                        }
                    }
                    else
                    {
                        if (input[i] == (25 + offset[12]))
                        {
                            vR[i] = 25;
                        }
                        else
                        {
                            vR[i] = 24;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (19 + offset[12]))
                {
                    if (input[i] > (21 + offset[12]))
                    {
                        if (input[i] == (23 + offset[12]))
                        {
                            vR[i] = 23;
                        }
                        else
                        {
                            vR[i] = 22;
                        }
                    }
                    else
                    {
                        if (input[i] == (21 + offset[12]))
                        {
                            vR[i] = 21;
                        }
                        else
                        {
                            vR[i] = 20;
                        }
                    }
                }
                else
                {
                    if (input[i] > (17 + offset[12]))
                    {
                        if (input[i] == (19 + offset[12]))
                        {
                            vR[i] = 19;
                        }
                        else
                        {
                            vR[i] = 18;
                        }
                    }
                    else
                    {
                        if (input[i] == (17 + offset[12]))
                        {
                            vR[i] = 17;
                        }
                        else
                        {
                            vR[i] = 16;
                        }
                    }
                }
            }
        }
        else
        {
            if (input[i] > (7 + offset[12]))
            {
                if (input[i] > (11 + offset[12]))
                {
                    if (input[i] > (13 + offset[12]))
                    {
                        if (input[i] == (15 + offset[12]))
                        {
                            vR[i] = 15;
                        }
                        else
                        {
                            vR[i] = 14;
                        }
                    }
                    else
                    {
                        if (input[i] == (13 + offset[12]))
                        {
                            vR[i] = 13;
                        }
                        else
                        {
                            vR[i] = 12;
                        }
                    }
                }
                else
                {
                    if (input[i] > (9 + offset[12]))
                    {
                        if (input[i] == (11 + offset[12]))
                        {
                            vR[i] = 11;
                        }
                        else
                        {
                            vR[i] = 10;
                        }
                    }
                    else
                    {
                        if (input[i] == (9 + offset[12]))
                        {
                            vR[i] = 9;
                        }
                        else
                        {
                            vR[i] = 8;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (3 + offset[12]))
                {
                    if (input[i] > (5 + offset[12]))
                    {
                        if (input[i] == (7 + offset[12]))
                        {
                            vR[i] = 7;
                        }
                        else
                        {
                            vR[i] = 6;
                        }
                    }
                    else
                    {
                        if (input[i] == (5 + offset[12]))
                        {
                            vR[i] = 5;
                        }
                        else
                        {
                            vR[i] = 4;
                        }
                    }
                }
                else
                {
                    if (input[i] > (1 + offset[12]))
                    {
                        if (input[i] == (3 + offset[12]))
                        {
                            vR[i] = 3;
                        }
                        else
                        {
                            vR[i] = 2;
                        }
                    }
                    else
                    {
                        if (input[i] == (1 + offset[12]))
                        {
                            vR[i] = 1;
                        }
                        else
                        {
                            vR[i] = 0;
                        }
                    }
                }
            }
        }
    }
    else if (threadIdx.x%WARP_SIZE == 13)
    {
        if (input[i] > (15 + offset[13]))
        {
            if (input[i] > (23 + offset[13]))
            {
                if (input[i] > (27 + offset[13]))
                {
                    if (input[i] > (29 + offset[13]))
                    {
                        if (input[i] == (31 + offset[13]))
                        {
                            vR[i] = 31;
                        }
                        else
                        {
                            vR[i] = 30;
                        }
                    }
                    else
                    {
                        if (input[i] == (29 + offset[13]))
                        {
                            vR[i] = 29;
                        }
                        else
                        {
                            vR[i] = 28;
                        }
                    }
                }
                else
                {
                    if (input[i] > (25 + offset[13]))
                    {
                        if (input[i] == (27 + offset[13]))
                        {
                            vR[i] = 27;
                        }
                        else
                        {
                            vR[i] = 26;
                        }
                    }
                    else
                    {
                        if (input[i] == (25 + offset[13]))
                        {
                            vR[i] = 25;
                        }
                        else
                        {
                            vR[i] = 24;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (19 + offset[13]))
                {
                    if (input[i] > (21 + offset[13]))
                    {
                        if (input[i] == (23 + offset[13]))
                        {
                            vR[i] = 23;
                        }
                        else
                        {
                            vR[i] = 22;
                        }
                    }
                    else
                    {
                        if (input[i] == (21 + offset[13]))
                        {
                            vR[i] = 21;
                        }
                        else
                        {
                            vR[i] = 20;
                        }
                    }
                }
                else
                {
                    if (input[i] > (17 + offset[13]))
                    {
                        if (input[i] == (19 + offset[13]))
                        {
                            vR[i] = 19;
                        }
                        else
                        {
                            vR[i] = 18;
                        }
                    }
                    else
                    {
                        if (input[i] == (17 + offset[13]))
                        {
                            vR[i] = 17;
                        }
                        else
                        {
                            vR[i] = 16;
                        }
                    }
                }
            }
        }
        else
        {
            if (input[i] > (7 + offset[13]))
            {
                if (input[i] > (11 + offset[13]))
                {
                    if (input[i] > (13 + offset[13]))
                    {
                        if (input[i] == (15 + offset[13]))
                        {
                            vR[i] = 15;
                        }
                        else
                        {
                            vR[i] = 14;
                        }
                    }
                    else
                    {
                        if (input[i] == (13 + offset[13]))
                        {
                            vR[i] = 13;
                        }
                        else
                        {
                            vR[i] = 12;
                        }
                    }
                }
                else
                {
                    if (input[i] > (9 + offset[13]))
                    {
                        if (input[i] == (11 + offset[13]))
                        {
                            vR[i] = 11;
                        }
                        else
                        {
                            vR[i] = 10;
                        }
                    }
                    else
                    {
                        if (input[i] == (9 + offset[13]))
                        {
                            vR[i] = 9;
                        }
                        else
                        {
                            vR[i] = 8;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (3 + offset[13]))
                {
                    if (input[i] > (5 + offset[13]))
                    {
                        if (input[i] == (7 + offset[13]))
                        {
                            vR[i] = 7;
                        }
                        else
                        {
                            vR[i] = 6;
                        }
                    }
                    else
                    {
                        if (input[i] == (5 + offset[13]))
                        {
                            vR[i] = 5;
                        }
                        else
                        {
                            vR[i] = 4;
                        }
                    }
                }
                else
                {
                    if (input[i] > (1 + offset[13]))
                    {
                        if (input[i] == (3 + offset[13]))
                        {
                            vR[i] = 3;
                        }
                        else
                        {
                            vR[i] = 2;
                        }
                    }
                    else
                    {
                        if (input[i] == (1 + offset[13]))
                        {
                            vR[i] = 1;
                        }
                        else
                        {
                            vR[i] = 0;
                        }
                    }
                }
            }
        }
    }
    else if (threadIdx.x%WARP_SIZE == 14)
    {
        if (input[i] > (15 + offset[14]))
        {
            if (input[i] > (23 + offset[14]))
            {
                if (input[i] > (27 + offset[14]))
                {
                    if (input[i] > (29 + offset[14]))
                    {
                        if (input[i] == (31 + offset[14]))
                        {
                            vR[i] = 31;
                        }
                        else
                        {
                            vR[i] = 30;
                        }
                    }
                    else
                    {
                        if (input[i] == (29 + offset[14]))
                        {
                            vR[i] = 29;
                        }
                        else
                        {
                            vR[i] = 28;
                        }
                    }
                }
                else
                {
                    if (input[i] > (25 + offset[14]))
                    {
                        if (input[i] == (27 + offset[14]))
                        {
                            vR[i] = 27;
                        }
                        else
                        {
                            vR[i] = 26;
                        }
                    }
                    else
                    {
                        if (input[i] == (25 + offset[14]))
                        {
                            vR[i] = 25;
                        }
                        else
                        {
                            vR[i] = 24;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (19 + offset[14]))
                {
                    if (input[i] > (21 + offset[14]))
                    {
                        if (input[i] == (23 + offset[14]))
                        {
                            vR[i] = 23;
                        }
                        else
                        {
                            vR[i] = 22;
                        }
                    }
                    else
                    {
                        if (input[i] == (21 + offset[14]))
                        {
                            vR[i] = 21;
                        }
                        else
                        {
                            vR[i] = 20;
                        }
                    }
                }
                else
                {
                    if (input[i] > (17 + offset[14]))
                    {
                        if (input[i] == (19 + offset[14]))
                        {
                            vR[i] = 19;
                        }
                        else
                        {
                            vR[i] = 18;
                        }
                    }
                    else
                    {
                        if (input[i] == (17 + offset[14]))
                        {
                            vR[i] = 17;
                        }
                        else
                        {
                            vR[i] = 16;
                        }
                    }
                }
            }
        }
        else
        {
            if (input[i] > (7 + offset[14]))
            {
                if (input[i] > (11 + offset[14]))
                {
                    if (input[i] > (13 + offset[14]))
                    {
                        if (input[i] == (15 + offset[14]))
                        {
                            vR[i] = 15;
                        }
                        else
                        {
                            vR[i] = 14;
                        }
                    }
                    else
                    {
                        if (input[i] == (13 + offset[14]))
                        {
                            vR[i] = 13;
                        }
                        else
                        {
                            vR[i] = 12;
                        }
                    }
                }
                else
                {
                    if (input[i] > (9 + offset[14]))
                    {
                        if (input[i] == (11 + offset[14]))
                        {
                            vR[i] = 11;
                        }
                        else
                        {
                            vR[i] = 10;
                        }
                    }
                    else
                    {
                        if (input[i] == (9 + offset[14]))
                        {
                            vR[i] = 9;
                        }
                        else
                        {
                            vR[i] = 8;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (3 + offset[14]))
                {
                    if (input[i] > (5 + offset[14]))
                    {
                        if (input[i] == (7 + offset[14]))
                        {
                            vR[i] = 7;
                        }
                        else
                        {
                            vR[i] = 6;
                        }
                    }
                    else
                    {
                        if (input[i] == (5 + offset[14]))
                        {
                            vR[i] = 5;
                        }
                        else
                        {
                            vR[i] = 4;
                        }
                    }
                }
                else
                {
                    if (input[i] > (1 + offset[14]))
                    {
                        if (input[i] == (3 + offset[14]))
                        {
                            vR[i] = 3;
                        }
                        else
                        {
                            vR[i] = 2;
                        }
                    }
                    else
                    {
                        if (input[i] == (1 + offset[14]))
                        {
                            vR[i] = 1;
                        }
                        else
                        {
                            vR[i] = 0;
                        }
                    }
                }
            }
        }
    }
    else if (threadIdx.x%WARP_SIZE == 15)
    {
        if (input[i] > (15 + offset[15]))
        {
            if (input[i] > (23 + offset[15]))
            {
                if (input[i] > (27 + offset[15]))
                {
                    if (input[i] > (29 + offset[15]))
                    {
                        if (input[i] == (31 + offset[15]))
                        {
                            vR[i] = 31;
                        }
                        else
                        {
                            vR[i] = 30;
                        }
                    }
                    else
                    {
                        if (input[i] == (29 + offset[15]))
                        {
                            vR[i] = 29;
                        }
                        else
                        {
                            vR[i] = 28;
                        }
                    }
                }
                else
                {
                    if (input[i] > (25 + offset[15]))
                    {
                        if (input[i] == (27 + offset[15]))
                        {
                            vR[i] = 27;
                        }
                        else
                        {
                            vR[i] = 26;
                        }
                    }
                    else
                    {
                        if (input[i] == (25 + offset[15]))
                        {
                            vR[i] = 25;
                        }
                        else
                        {
                            vR[i] = 24;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (19 + offset[15]))
                {
                    if (input[i] > (21 + offset[15]))
                    {
                        if (input[i] == (23 + offset[15]))
                        {
                            vR[i] = 23;
                        }
                        else
                        {
                            vR[i] = 22;
                        }
                    }
                    else
                    {
                        if (input[i] == (21 + offset[15]))
                        {
                            vR[i] = 21;
                        }
                        else
                        {
                            vR[i] = 20;
                        }
                    }
                }
                else
                {
                    if (input[i] > (17 + offset[15]))
                    {
                        if (input[i] == (19 + offset[15]))
                        {
                            vR[i] = 19;
                        }
                        else
                        {
                            vR[i] = 18;
                        }
                    }
                    else
                    {
                        if (input[i] == (17 + offset[15]))
                        {
                            vR[i] = 17;
                        }
                        else
                        {
                            vR[i] = 16;
                        }
                    }
                }
            }
        }
        else
        {
            if (input[i] > (7 + offset[15]))
            {
                if (input[i] > (11 + offset[15]))
                {
                    if (input[i] > (13 + offset[15]))
                    {
                        if (input[i] == (15 + offset[15]))
                        {
                            vR[i] = 15;
                        }
                        else
                        {
                            vR[i] = 14;
                        }
                    }
                    else
                    {
                        if (input[i] == (13 + offset[15]))
                        {
                            vR[i] = 13;
                        }
                        else
                        {
                            vR[i] = 12;
                        }
                    }
                }
                else
                {
                    if (input[i] > (9 + offset[15]))
                    {
                        if (input[i] == (11 + offset[15]))
                        {
                            vR[i] = 11;
                        }
                        else
                        {
                            vR[i] = 10;
                        }
                    }
                    else
                    {
                        if (input[i] == (9 + offset[15]))
                        {
                            vR[i] = 9;
                        }
                        else
                        {
                            vR[i] = 8;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (3 + offset[15]))
                {
                    if (input[i] > (5 + offset[15]))
                    {
                        if (input[i] == (7 + offset[15]))
                        {
                            vR[i] = 7;
                        }
                        else
                        {
                            vR[i] = 6;
                        }
                    }
                    else
                    {
                        if (input[i] == (5 + offset[15]))
                        {
                            vR[i] = 5;
                        }
                        else
                        {
                            vR[i] = 4;
                        }
                    }
                }
                else
                {
                    if (input[i] > (1 + offset[15]))
                    {
                        if (input[i] == (3 + offset[15]))
                        {
                            vR[i] = 3;
                        }
                        else
                        {
                            vR[i] = 2;
                        }
                    }
                    else
                    {
                        if (input[i] == (1 + offset[15]))
                        {
                            vR[i] = 1;
                        }
                        else
                        {
                            vR[i] = 0;
                        }
                    }
                }
            }
        }
    }
    else if (threadIdx.x%WARP_SIZE == 16)
    {
        if (input[i] > (15 + offset[16]))
        {
            if (input[i] > (23 + offset[16]))
            {
                if (input[i] > (27 + offset[16]))
                {
                    if (input[i] > (29 + offset[16]))
                    {
                        if (input[i] == (31 + offset[16]))
                        {
                            vR[i] = 31;
                        }
                        else
                        {
                            vR[i] = 30;
                        }
                    }
                    else
                    {
                        if (input[i] == (29 + offset[16]))
                        {
                            vR[i] = 29;
                        }
                        else
                        {
                            vR[i] = 28;
                        }
                    }
                }
                else
                {
                    if (input[i] > (25 + offset[16]))
                    {
                        if (input[i] == (27 + offset[16]))
                        {
                            vR[i] = 27;
                        }
                        else
                        {
                            vR[i] = 26;
                        }
                    }
                    else
                    {
                        if (input[i] == (25 + offset[16]))
                        {
                            vR[i] = 25;
                        }
                        else
                        {
                            vR[i] = 24;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (19 + offset[16]))
                {
                    if (input[i] > (21 + offset[16]))
                    {
                        if (input[i] == (23 + offset[16]))
                        {
                            vR[i] = 23;
                        }
                        else
                        {
                            vR[i] = 22;
                        }
                    }
                    else
                    {
                        if (input[i] == (21 + offset[16]))
                        {
                            vR[i] = 21;
                        }
                        else
                        {
                            vR[i] = 20;
                        }
                    }
                }
                else
                {
                    if (input[i] > (17 + offset[16]))
                    {
                        if (input[i] == (19 + offset[16]))
                        {
                            vR[i] = 19;
                        }
                        else
                        {
                            vR[i] = 18;
                        }
                    }
                    else
                    {
                        if (input[i] == (17 + offset[16]))
                        {
                            vR[i] = 17;
                        }
                        else
                        {
                            vR[i] = 16;
                        }
                    }
                }
            }
        }
        else
        {
            if (input[i] > (7 + offset[16]))
            {
                if (input[i] > (11 + offset[16]))
                {
                    if (input[i] > (13 + offset[16]))
                    {
                        if (input[i] == (15 + offset[16]))
                        {
                            vR[i] = 15;
                        }
                        else
                        {
                            vR[i] = 14;
                        }
                    }
                    else
                    {
                        if (input[i] == (13 + offset[16]))
                        {
                            vR[i] = 13;
                        }
                        else
                        {
                            vR[i] = 12;
                        }
                    }
                }
                else
                {
                    if (input[i] > (9 + offset[16]))
                    {
                        if (input[i] == (11 + offset[16]))
                        {
                            vR[i] = 11;
                        }
                        else
                        {
                            vR[i] = 10;
                        }
                    }
                    else
                    {
                        if (input[i] == (9 + offset[16]))
                        {
                            vR[i] = 9;
                        }
                        else
                        {
                            vR[i] = 8;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (3 + offset[16]))
                {
                    if (input[i] > (5 + offset[16]))
                    {
                        if (input[i] == (7 + offset[16]))
                        {
                            vR[i] = 7;
                        }
                        else
                        {
                            vR[i] = 6;
                        }
                    }
                    else
                    {
                        if (input[i] == (5 + offset[16]))
                        {
                            vR[i] = 5;
                        }
                        else
                        {
                            vR[i] = 4;
                        }
                    }
                }
                else
                {
                    if (input[i] > (1 + offset[16]))
                    {
                        if (input[i] == (3 + offset[16]))
                        {
                            vR[i] = 3;
                        }
                        else
                        {
                            vR[i] = 2;
                        }
                    }
                    else
                    {
                        if (input[i] == (1 + offset[16]))
                        {
                            vR[i] = 1;
                        }
                        else
                        {
                            vR[i] = 0;
                        }
                    }
                }
            }
        }
    }
    else if (threadIdx.x%WARP_SIZE == 17)
    {
        if (input[i] > (15 + offset[17]))
        {
            if (input[i] > (23 + offset[17]))
            {
                if (input[i] > (27 + offset[17]))
                {
                    if (input[i] > (29 + offset[17]))
                    {
                        if (input[i] == (31 + offset[17]))
                        {
                            vR[i] = 31;
                        }
                        else
                        {
                            vR[i] = 30;
                        }
                    }
                    else
                    {
                        if (input[i] == (29 + offset[17]))
                        {
                            vR[i] = 29;
                        }
                        else
                        {
                            vR[i] = 28;
                        }
                    }
                }
                else
                {
                    if (input[i] > (25 + offset[17]))
                    {
                        if (input[i] == (27 + offset[17]))
                        {
                            vR[i] = 27;
                        }
                        else
                        {
                            vR[i] = 26;
                        }
                    }
                    else
                    {
                        if (input[i] == (25 + offset[17]))
                        {
                            vR[i] = 25;
                        }
                        else
                        {
                            vR[i] = 24;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (19 + offset[17]))
                {
                    if (input[i] > (21 + offset[17]))
                    {
                        if (input[i] == (23 + offset[17]))
                        {
                            vR[i] = 23;
                        }
                        else
                        {
                            vR[i] = 22;
                        }
                    }
                    else
                    {
                        if (input[i] == (21 + offset[17]))
                        {
                            vR[i] = 21;
                        }
                        else
                        {
                            vR[i] = 20;
                        }
                    }
                }
                else
                {
                    if (input[i] > (17 + offset[17]))
                    {
                        if (input[i] == (19 + offset[17]))
                        {
                            vR[i] = 19;
                        }
                        else
                        {
                            vR[i] = 18;
                        }
                    }
                    else
                    {
                        if (input[i] == (17 + offset[17]))
                        {
                            vR[i] = 17;
                        }
                        else
                        {
                            vR[i] = 16;
                        }
                    }
                }
            }
        }
        else
        {
            if (input[i] > (7 + offset[17]))
            {
                if (input[i] > (11 + offset[17]))
                {
                    if (input[i] > (13 + offset[17]))
                    {
                        if (input[i] == (15 + offset[17]))
                        {
                            vR[i] = 15;
                        }
                        else
                        {
                            vR[i] = 14;
                        }
                    }
                    else
                    {
                        if (input[i] == (13 + offset[17]))
                        {
                            vR[i] = 13;
                        }
                        else
                        {
                            vR[i] = 12;
                        }
                    }
                }
                else
                {
                    if (input[i] > (9 + offset[17]))
                    {
                        if (input[i] == (11 + offset[17]))
                        {
                            vR[i] = 11;
                        }
                        else
                        {
                            vR[i] = 10;
                        }
                    }
                    else
                    {
                        if (input[i] == (9 + offset[17]))
                        {
                            vR[i] = 9;
                        }
                        else
                        {
                            vR[i] = 8;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (3 + offset[17]))
                {
                    if (input[i] > (5 + offset[17]))
                    {
                        if (input[i] == (7 + offset[17]))
                        {
                            vR[i] = 7;
                        }
                        else
                        {
                            vR[i] = 6;
                        }
                    }
                    else
                    {
                        if (input[i] == (5 + offset[17]))
                        {
                            vR[i] = 5;
                        }
                        else
                        {
                            vR[i] = 4;
                        }
                    }
                }
                else
                {
                    if (input[i] > (1 + offset[17]))
                    {
                        if (input[i] == (3 + offset[17]))
                        {
                            vR[i] = 3;
                        }
                        else
                        {
                            vR[i] = 2;
                        }
                    }
                    else
                    {
                        if (input[i] == (1 + offset[17]))
                        {
                            vR[i] = 1;
                        }
                        else
                        {
                            vR[i] = 0;
                        }
                    }
                }
            }
        }
    }
    else if (threadIdx.x%WARP_SIZE == 18)
    {
        if (input[i] > (15 + offset[18]))
        {
            if (input[i] > (23 + offset[18]))
            {
                if (input[i] > (27 + offset[18]))
                {
                    if (input[i] > (29 + offset[18]))
                    {
                        if (input[i] == (31 + offset[18]))
                        {
                            vR[i] = 31;
                        }
                        else
                        {
                            vR[i] = 30;
                        }
                    }
                    else
                    {
                        if (input[i] == (29 + offset[18]))
                        {
                            vR[i] = 29;
                        }
                        else
                        {
                            vR[i] = 28;
                        }
                    }
                }
                else
                {
                    if (input[i] > (25 + offset[18]))
                    {
                        if (input[i] == (27 + offset[18]))
                        {
                            vR[i] = 27;
                        }
                        else
                        {
                            vR[i] = 26;
                        }
                    }
                    else
                    {
                        if (input[i] == (25 + offset[18]))
                        {
                            vR[i] = 25;
                        }
                        else
                        {
                            vR[i] = 24;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (19 + offset[18]))
                {
                    if (input[i] > (21 + offset[18]))
                    {
                        if (input[i] == (23 + offset[18]))
                        {
                            vR[i] = 23;
                        }
                        else
                        {
                            vR[i] = 22;
                        }
                    }
                    else
                    {
                        if (input[i] == (21 + offset[18]))
                        {
                            vR[i] = 21;
                        }
                        else
                        {
                            vR[i] = 20;
                        }
                    }
                }
                else
                {
                    if (input[i] > (17 + offset[18]))
                    {
                        if (input[i] == (19 + offset[18]))
                        {
                            vR[i] = 19;
                        }
                        else
                        {
                            vR[i] = 18;
                        }
                    }
                    else
                    {
                        if (input[i] == (17 + offset[18]))
                        {
                            vR[i] = 17;
                        }
                        else
                        {
                            vR[i] = 16;
                        }
                    }
                }
            }
        }
        else
        {
            if (input[i] > (7 + offset[18]))
            {
                if (input[i] > (11 + offset[18]))
                {
                    if (input[i] > (13 + offset[18]))
                    {
                        if (input[i] == (15 + offset[18]))
                        {
                            vR[i] = 15;
                        }
                        else
                        {
                            vR[i] = 14;
                        }
                    }
                    else
                    {
                        if (input[i] == (13 + offset[18]))
                        {
                            vR[i] = 13;
                        }
                        else
                        {
                            vR[i] = 12;
                        }
                    }
                }
                else
                {
                    if (input[i] > (9 + offset[18]))
                    {
                        if (input[i] == (11 + offset[18]))
                        {
                            vR[i] = 11;
                        }
                        else
                        {
                            vR[i] = 10;
                        }
                    }
                    else
                    {
                        if (input[i] == (9 + offset[18]))
                        {
                            vR[i] = 9;
                        }
                        else
                        {
                            vR[i] = 8;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (3 + offset[18]))
                {
                    if (input[i] > (5 + offset[18]))
                    {
                        if (input[i] == (7 + offset[18]))
                        {
                            vR[i] = 7;
                        }
                        else
                        {
                            vR[i] = 6;
                        }
                    }
                    else
                    {
                        if (input[i] == (5 + offset[18]))
                        {
                            vR[i] = 5;
                        }
                        else
                        {
                            vR[i] = 4;
                        }
                    }
                }
                else
                {
                    if (input[i] > (1 + offset[18]))
                    {
                        if (input[i] == (3 + offset[18]))
                        {
                            vR[i] = 3;
                        }
                        else
                        {
                            vR[i] = 2;
                        }
                    }
                    else
                    {
                        if (input[i] == (1 + offset[18]))
                        {
                            vR[i] = 1;
                        }
                        else
                        {
                            vR[i] = 0;
                        }
                    }
                }
            }
        }
    }
    else if (threadIdx.x%WARP_SIZE == 19)
    {
        if (input[i] > (15 + offset[19]))
        {
            if (input[i] > (23 + offset[19]))
            {
                if (input[i] > (27 + offset[19]))
                {
                    if (input[i] > (29 + offset[19]))
                    {
                        if (input[i] == (31 + offset[19]))
                        {
                            vR[i] = 31;
                        }
                        else
                        {
                            vR[i] = 30;
                        }
                    }
                    else
                    {
                        if (input[i] == (29 + offset[19]))
                        {
                            vR[i] = 29;
                        }
                        else
                        {
                            vR[i] = 28;
                        }
                    }
                }
                else
                {
                    if (input[i] > (25 + offset[19]))
                    {
                        if (input[i] == (27 + offset[19]))
                        {
                            vR[i] = 27;
                        }
                        else
                        {
                            vR[i] = 26;
                        }
                    }
                    else
                    {
                        if (input[i] == (25 + offset[19]))
                        {
                            vR[i] = 25;
                        }
                        else
                        {
                            vR[i] = 24;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (19 + offset[19]))
                {
                    if (input[i] > (21 + offset[19]))
                    {
                        if (input[i] == (23 + offset[19]))
                        {
                            vR[i] = 23;
                        }
                        else
                        {
                            vR[i] = 22;
                        }
                    }
                    else
                    {
                        if (input[i] == (21 + offset[19]))
                        {
                            vR[i] = 21;
                        }
                        else
                        {
                            vR[i] = 20;
                        }
                    }
                }
                else
                {
                    if (input[i] > (17 + offset[19]))
                    {
                        if (input[i] == (19 + offset[19]))
                        {
                            vR[i] = 19;
                        }
                        else
                        {
                            vR[i] = 18;
                        }
                    }
                    else
                    {
                        if (input[i] == (17 + offset[19]))
                        {
                            vR[i] = 17;
                        }
                        else
                        {
                            vR[i] = 16;
                        }
                    }
                }
            }
        }
        else
        {
            if (input[i] > (7 + offset[19]))
            {
                if (input[i] > (11 + offset[19]))
                {
                    if (input[i] > (13 + offset[19]))
                    {
                        if (input[i] == (15 + offset[19]))
                        {
                            vR[i] = 15;
                        }
                        else
                        {
                            vR[i] = 14;
                        }
                    }
                    else
                    {
                        if (input[i] == (13 + offset[19]))
                        {
                            vR[i] = 13;
                        }
                        else
                        {
                            vR[i] = 12;
                        }
                    }
                }
                else
                {
                    if (input[i] > (9 + offset[19]))
                    {
                        if (input[i] == (11 + offset[19]))
                        {
                            vR[i] = 11;
                        }
                        else
                        {
                            vR[i] = 10;
                        }
                    }
                    else
                    {
                        if (input[i] == (9 + offset[19]))
                        {
                            vR[i] = 9;
                        }
                        else
                        {
                            vR[i] = 8;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (3 + offset[19]))
                {
                    if (input[i] > (5 + offset[19]))
                    {
                        if (input[i] == (7 + offset[19]))
                        {
                            vR[i] = 7;
                        }
                        else
                        {
                            vR[i] = 6;
                        }
                    }
                    else
                    {
                        if (input[i] == (5 + offset[19]))
                        {
                            vR[i] = 5;
                        }
                        else
                        {
                            vR[i] = 4;
                        }
                    }
                }
                else
                {
                    if (input[i] > (1 + offset[19]))
                    {
                        if (input[i] == (3 + offset[19]))
                        {
                            vR[i] = 3;
                        }
                        else
                        {
                            vR[i] = 2;
                        }
                    }
                    else
                    {
                        if (input[i] == (1 + offset[19]))
                        {
                            vR[i] = 1;
                        }
                        else
                        {
                            vR[i] = 0;
                        }
                    }
                }
            }
        }
    }
    else if (threadIdx.x%WARP_SIZE == 20)
    {
        if (input[i] > (15 + offset[20]))
        {
            if (input[i] > (23 + offset[20]))
            {
                if (input[i] > (27 + offset[20]))
                {
                    if (input[i] > (29 + offset[20]))
                    {
                        if (input[i] == (31 + offset[20]))
                        {
                            vR[i] = 31;
                        }
                        else
                        {
                            vR[i] = 30;
                        }
                    }
                    else
                    {
                        if (input[i] == (29 + offset[20]))
                        {
                            vR[i] = 29;
                        }
                        else
                        {
                            vR[i] = 28;
                        }
                    }
                }
                else
                {
                    if (input[i] > (25 + offset[20]))
                    {
                        if (input[i] == (27 + offset[20]))
                        {
                            vR[i] = 27;
                        }
                        else
                        {
                            vR[i] = 26;
                        }
                    }
                    else
                    {
                        if (input[i] == (25 + offset[20]))
                        {
                            vR[i] = 25;
                        }
                        else
                        {
                            vR[i] = 24;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (19 + offset[20]))
                {
                    if (input[i] > (21 + offset[20]))
                    {
                        if (input[i] == (23 + offset[20]))
                        {
                            vR[i] = 23;
                        }
                        else
                        {
                            vR[i] = 22;
                        }
                    }
                    else
                    {
                        if (input[i] == (21 + offset[20]))
                        {
                            vR[i] = 21;
                        }
                        else
                        {
                            vR[i] = 20;
                        }
                    }
                }
                else
                {
                    if (input[i] > (17 + offset[20]))
                    {
                        if (input[i] == (19 + offset[20]))
                        {
                            vR[i] = 19;
                        }
                        else
                        {
                            vR[i] = 18;
                        }
                    }
                    else
                    {
                        if (input[i] == (17 + offset[20]))
                        {
                            vR[i] = 17;
                        }
                        else
                        {
                            vR[i] = 16;
                        }
                    }
                }
            }
        }
        else
        {
            if (input[i] > (7 + offset[20]))
            {
                if (input[i] > (11 + offset[20]))
                {
                    if (input[i] > (13 + offset[20]))
                    {
                        if (input[i] == (15 + offset[20]))
                        {
                            vR[i] = 15;
                        }
                        else
                        {
                            vR[i] = 14;
                        }
                    }
                    else
                    {
                        if (input[i] == (13 + offset[20]))
                        {
                            vR[i] = 13;
                        }
                        else
                        {
                            vR[i] = 12;
                        }
                    }
                }
                else
                {
                    if (input[i] > (9 + offset[20]))
                    {
                        if (input[i] == (11 + offset[20]))
                        {
                            vR[i] = 11;
                        }
                        else
                        {
                            vR[i] = 10;
                        }
                    }
                    else
                    {
                        if (input[i] == (9 + offset[20]))
                        {
                            vR[i] = 9;
                        }
                        else
                        {
                            vR[i] = 8;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (3 + offset[20]))
                {
                    if (input[i] > (5 + offset[20]))
                    {
                        if (input[i] == (7 + offset[20]))
                        {
                            vR[i] = 7;
                        }
                        else
                        {
                            vR[i] = 6;
                        }
                    }
                    else
                    {
                        if (input[i] == (5 + offset[20]))
                        {
                            vR[i] = 5;
                        }
                        else
                        {
                            vR[i] = 4;
                        }
                    }
                }
                else
                {
                    if (input[i] > (1 + offset[20]))
                    {
                        if (input[i] == (3 + offset[20]))
                        {
                            vR[i] = 3;
                        }
                        else
                        {
                            vR[i] = 2;
                        }
                    }
                    else
                    {
                        if (input[i] == (1 + offset[20]))
                        {
                            vR[i] = 1;
                        }
                        else
                        {
                            vR[i] = 0;
                        }
                    }
                }
            }
        }
    }
    else if (threadIdx.x%WARP_SIZE == 21)
    {
        if (input[i] > (15 + offset[21]))
        {
            if (input[i] > (23 + offset[21]))
            {
                if (input[i] > (27 + offset[21]))
                {
                    if (input[i] > (29 + offset[21]))
                    {
                        if (input[i] == (31 + offset[21]))
                        {
                            vR[i] = 31;
                        }
                        else
                        {
                            vR[i] = 30;
                        }
                    }
                    else
                    {
                        if (input[i] == (29 + offset[21]))
                        {
                            vR[i] = 29;
                        }
                        else
                        {
                            vR[i] = 28;
                        }
                    }
                }
                else
                {
                    if (input[i] > (25 + offset[21]))
                    {
                        if (input[i] == (27 + offset[21]))
                        {
                            vR[i] = 27;
                        }
                        else
                        {
                            vR[i] = 26;
                        }
                    }
                    else
                    {
                        if (input[i] == (25 + offset[21]))
                        {
                            vR[i] = 25;
                        }
                        else
                        {
                            vR[i] = 24;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (19 + offset[21]))
                {
                    if (input[i] > (21 + offset[21]))
                    {
                        if (input[i] == (23 + offset[21]))
                        {
                            vR[i] = 23;
                        }
                        else
                        {
                            vR[i] = 22;
                        }
                    }
                    else
                    {
                        if (input[i] == (21 + offset[21]))
                        {
                            vR[i] = 21;
                        }
                        else
                        {
                            vR[i] = 20;
                        }
                    }
                }
                else
                {
                    if (input[i] > (17 + offset[21]))
                    {
                        if (input[i] == (19 + offset[21]))
                        {
                            vR[i] = 19;
                        }
                        else
                        {
                            vR[i] = 18;
                        }
                    }
                    else
                    {
                        if (input[i] == (17 + offset[21]))
                        {
                            vR[i] = 17;
                        }
                        else
                        {
                            vR[i] = 16;
                        }
                    }
                }
            }
        }
        else
        {
            if (input[i] > (7 + offset[21]))
            {
                if (input[i] > (11 + offset[21]))
                {
                    if (input[i] > (13 + offset[21]))
                    {
                        if (input[i] == (15 + offset[21]))
                        {
                            vR[i] = 15;
                        }
                        else
                        {
                            vR[i] = 14;
                        }
                    }
                    else
                    {
                        if (input[i] == (13 + offset[21]))
                        {
                            vR[i] = 13;
                        }
                        else
                        {
                            vR[i] = 12;
                        }
                    }
                }
                else
                {
                    if (input[i] > (9 + offset[21]))
                    {
                        if (input[i] == (11 + offset[21]))
                        {
                            vR[i] = 11;
                        }
                        else
                        {
                            vR[i] = 10;
                        }
                    }
                    else
                    {
                        if (input[i] == (9 + offset[21]))
                        {
                            vR[i] = 9;
                        }
                        else
                        {
                            vR[i] = 8;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (3 + offset[21]))
                {
                    if (input[i] > (5 + offset[21]))
                    {
                        if (input[i] == (7 + offset[21]))
                        {
                            vR[i] = 7;
                        }
                        else
                        {
                            vR[i] = 6;
                        }
                    }
                    else
                    {
                        if (input[i] == (5 + offset[21]))
                        {
                            vR[i] = 5;
                        }
                        else
                        {
                            vR[i] = 4;
                        }
                    }
                }
                else
                {
                    if (input[i] > (1 + offset[21]))
                    {
                        if (input[i] == (3 + offset[21]))
                        {
                            vR[i] = 3;
                        }
                        else
                        {
                            vR[i] = 2;
                        }
                    }
                    else
                    {
                        if (input[i] == (1 + offset[21]))
                        {
                            vR[i] = 1;
                        }
                        else
                        {
                            vR[i] = 0;
                        }
                    }
                }
            }
        }
    }
    else if (threadIdx.x%WARP_SIZE == 22)
    {
        if (input[i] > (15 + offset[22]))
        {
            if (input[i] > (23 + offset[22]))
            {
                if (input[i] > (27 + offset[22]))
                {
                    if (input[i] > (29 + offset[22]))
                    {
                        if (input[i] == (31 + offset[22]))
                        {
                            vR[i] = 31;
                        }
                        else
                        {
                            vR[i] = 30;
                        }
                    }
                    else
                    {
                        if (input[i] == (29 + offset[22]))
                        {
                            vR[i] = 29;
                        }
                        else
                        {
                            vR[i] = 28;
                        }
                    }
                }
                else
                {
                    if (input[i] > (25 + offset[22]))
                    {
                        if (input[i] == (27 + offset[22]))
                        {
                            vR[i] = 27;
                        }
                        else
                        {
                            vR[i] = 26;
                        }
                    }
                    else
                    {
                        if (input[i] == (25 + offset[22]))
                        {
                            vR[i] = 25;
                        }
                        else
                        {
                            vR[i] = 24;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (19 + offset[22]))
                {
                    if (input[i] > (21 + offset[22]))
                    {
                        if (input[i] == (23 + offset[22]))
                        {
                            vR[i] = 23;
                        }
                        else
                        {
                            vR[i] = 22;
                        }
                    }
                    else
                    {
                        if (input[i] == (21 + offset[22]))
                        {
                            vR[i] = 21;
                        }
                        else
                        {
                            vR[i] = 20;
                        }
                    }
                }
                else
                {
                    if (input[i] > (17 + offset[22]))
                    {
                        if (input[i] == (19 + offset[22]))
                        {
                            vR[i] = 19;
                        }
                        else
                        {
                            vR[i] = 18;
                        }
                    }
                    else
                    {
                        if (input[i] == (17 + offset[22]))
                        {
                            vR[i] = 17;
                        }
                        else
                        {
                            vR[i] = 16;
                        }
                    }
                }
            }
        }
        else
        {
            if (input[i] > (7 + offset[22]))
            {
                if (input[i] > (11 + offset[22]))
                {
                    if (input[i] > (13 + offset[22]))
                    {
                        if (input[i] == (15 + offset[22]))
                        {
                            vR[i] = 15;
                        }
                        else
                        {
                            vR[i] = 14;
                        }
                    }
                    else
                    {
                        if (input[i] == (13 + offset[22]))
                        {
                            vR[i] = 13;
                        }
                        else
                        {
                            vR[i] = 12;
                        }
                    }
                }
                else
                {
                    if (input[i] > (9 + offset[22]))
                    {
                        if (input[i] == (11 + offset[22]))
                        {
                            vR[i] = 11;
                        }
                        else
                        {
                            vR[i] = 10;
                        }
                    }
                    else
                    {
                        if (input[i] == (9 + offset[22]))
                        {
                            vR[i] = 9;
                        }
                        else
                        {
                            vR[i] = 8;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (3 + offset[22]))
                {
                    if (input[i] > (5 + offset[22]))
                    {
                        if (input[i] == (7 + offset[22]))
                        {
                            vR[i] = 7;
                        }
                        else
                        {
                            vR[i] = 6;
                        }
                    }
                    else
                    {
                        if (input[i] == (5 + offset[22]))
                        {
                            vR[i] = 5;
                        }
                        else
                        {
                            vR[i] = 4;
                        }
                    }
                }
                else
                {
                    if (input[i] > (1 + offset[22]))
                    {
                        if (input[i] == (3 + offset[22]))
                        {
                            vR[i] = 3;
                        }
                        else
                        {
                            vR[i] = 2;
                        }
                    }
                    else
                    {
                        if (input[i] == (1 + offset[22]))
                        {
                            vR[i] = 1;
                        }
                        else
                        {
                            vR[i] = 0;
                        }
                    }
                }
            }
        }
    }
    else if (threadIdx.x%WARP_SIZE == 23)
    {
        if (input[i] > (15 + offset[23]))
        {
            if (input[i] > (23 + offset[23]))
            {
                if (input[i] > (27 + offset[23]))
                {
                    if (input[i] > (29 + offset[23]))
                    {
                        if (input[i] == (31 + offset[23]))
                        {
                            vR[i] = 31;
                        }
                        else
                        {
                            vR[i] = 30;
                        }
                    }
                    else
                    {
                        if (input[i] == (29 + offset[23]))
                        {
                            vR[i] = 29;
                        }
                        else
                        {
                            vR[i] = 28;
                        }
                    }
                }
                else
                {
                    if (input[i] > (25 + offset[23]))
                    {
                        if (input[i] == (27 + offset[23]))
                        {
                            vR[i] = 27;
                        }
                        else
                        {
                            vR[i] = 26;
                        }
                    }
                    else
                    {
                        if (input[i] == (25 + offset[23]))
                        {
                            vR[i] = 25;
                        }
                        else
                        {
                            vR[i] = 24;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (19 + offset[23]))
                {
                    if (input[i] > (21 + offset[23]))
                    {
                        if (input[i] == (23 + offset[23]))
                        {
                            vR[i] = 23;
                        }
                        else
                        {
                            vR[i] = 22;
                        }
                    }
                    else
                    {
                        if (input[i] == (21 + offset[23]))
                        {
                            vR[i] = 21;
                        }
                        else
                        {
                            vR[i] = 20;
                        }
                    }
                }
                else
                {
                    if (input[i] > (17 + offset[23]))
                    {
                        if (input[i] == (19 + offset[23]))
                        {
                            vR[i] = 19;
                        }
                        else
                        {
                            vR[i] = 18;
                        }
                    }
                    else
                    {
                        if (input[i] == (17 + offset[23]))
                        {
                            vR[i] = 17;
                        }
                        else
                        {
                            vR[i] = 16;
                        }
                    }
                }
            }
        }
        else
        {
            if (input[i] > (7 + offset[23]))
            {
                if (input[i] > (11 + offset[23]))
                {
                    if (input[i] > (13 + offset[23]))
                    {
                        if (input[i] == (15 + offset[23]))
                        {
                            vR[i] = 15;
                        }
                        else
                        {
                            vR[i] = 14;
                        }
                    }
                    else
                    {
                        if (input[i] == (13 + offset[23]))
                        {
                            vR[i] = 13;
                        }
                        else
                        {
                            vR[i] = 12;
                        }
                    }
                }
                else
                {
                    if (input[i] > (9 + offset[23]))
                    {
                        if (input[i] == (11 + offset[23]))
                        {
                            vR[i] = 11;
                        }
                        else
                        {
                            vR[i] = 10;
                        }
                    }
                    else
                    {
                        if (input[i] == (9 + offset[23]))
                        {
                            vR[i] = 9;
                        }
                        else
                        {
                            vR[i] = 8;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (3 + offset[23]))
                {
                    if (input[i] > (5 + offset[23]))
                    {
                        if (input[i] == (7 + offset[23]))
                        {
                            vR[i] = 7;
                        }
                        else
                        {
                            vR[i] = 6;
                        }
                    }
                    else
                    {
                        if (input[i] == (5 + offset[23]))
                        {
                            vR[i] = 5;
                        }
                        else
                        {
                            vR[i] = 4;
                        }
                    }
                }
                else
                {
                    if (input[i] > (1 + offset[23]))
                    {
                        if (input[i] == (3 + offset[23]))
                        {
                            vR[i] = 3;
                        }
                        else
                        {
                            vR[i] = 2;
                        }
                    }
                    else
                    {
                        if (input[i] == (1 + offset[23]))
                        {
                            vR[i] = 1;
                        }
                        else
                        {
                            vR[i] = 0;
                        }
                    }
                }
            }
        }
    }
    else if (threadIdx.x%WARP_SIZE == 24)
    {
        if (input[i] > (15 + offset[24]))
        {
            if (input[i] > (23 + offset[24]))
            {
                if (input[i] > (27 + offset[24]))
                {
                    if (input[i] > (29 + offset[24]))
                    {
                        if (input[i] == (31 + offset[24]))
                        {
                            vR[i] = 31;
                        }
                        else
                        {
                            vR[i] = 30;
                        }
                    }
                    else
                    {
                        if (input[i] == (29 + offset[24]))
                        {
                            vR[i] = 29;
                        }
                        else
                        {
                            vR[i] = 28;
                        }
                    }
                }
                else
                {
                    if (input[i] > (25 + offset[24]))
                    {
                        if (input[i] == (27 + offset[24]))
                        {
                            vR[i] = 27;
                        }
                        else
                        {
                            vR[i] = 26;
                        }
                    }
                    else
                    {
                        if (input[i] == (25 + offset[24]))
                        {
                            vR[i] = 25;
                        }
                        else
                        {
                            vR[i] = 24;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (19 + offset[24]))
                {
                    if (input[i] > (21 + offset[24]))
                    {
                        if (input[i] == (23 + offset[24]))
                        {
                            vR[i] = 23;
                        }
                        else
                        {
                            vR[i] = 22;
                        }
                    }
                    else
                    {
                        if (input[i] == (21 + offset[24]))
                        {
                            vR[i] = 21;
                        }
                        else
                        {
                            vR[i] = 20;
                        }
                    }
                }
                else
                {
                    if (input[i] > (17 + offset[24]))
                    {
                        if (input[i] == (19 + offset[24]))
                        {
                            vR[i] = 19;
                        }
                        else
                        {
                            vR[i] = 18;
                        }
                    }
                    else
                    {
                        if (input[i] == (17 + offset[24]))
                        {
                            vR[i] = 17;
                        }
                        else
                        {
                            vR[i] = 16;
                        }
                    }
                }
            }
        }
        else
        {
            if (input[i] > (7 + offset[24]))
            {
                if (input[i] > (11 + offset[24]))
                {
                    if (input[i] > (13 + offset[24]))
                    {
                        if (input[i] == (15 + offset[24]))
                        {
                            vR[i] = 15;
                        }
                        else
                        {
                            vR[i] = 14;
                        }
                    }
                    else
                    {
                        if (input[i] == (13 + offset[24]))
                        {
                            vR[i] = 13;
                        }
                        else
                        {
                            vR[i] = 12;
                        }
                    }
                }
                else
                {
                    if (input[i] > (9 + offset[24]))
                    {
                        if (input[i] == (11 + offset[24]))
                        {
                            vR[i] = 11;
                        }
                        else
                        {
                            vR[i] = 10;
                        }
                    }
                    else
                    {
                        if (input[i] == (9 + offset[24]))
                        {
                            vR[i] = 9;
                        }
                        else
                        {
                            vR[i] = 8;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (3 + offset[24]))
                {
                    if (input[i] > (5 + offset[24]))
                    {
                        if (input[i] == (7 + offset[24]))
                        {
                            vR[i] = 7;
                        }
                        else
                        {
                            vR[i] = 6;
                        }
                    }
                    else
                    {
                        if (input[i] == (5 + offset[24]))
                        {
                            vR[i] = 5;
                        }
                        else
                        {
                            vR[i] = 4;
                        }
                    }
                }
                else
                {
                    if (input[i] > (1 + offset[24]))
                    {
                        if (input[i] == (3 + offset[24]))
                        {
                            vR[i] = 3;
                        }
                        else
                        {
                            vR[i] = 2;
                        }
                    }
                    else
                    {
                        if (input[i] == (1 + offset[24]))
                        {
                            vR[i] = 1;
                        }
                        else
                        {
                            vR[i] = 0;
                        }
                    }
                }
            }
        }
    }
    else if (threadIdx.x%WARP_SIZE == 25)
    {
        if (input[i] > (15 + offset[25]))
        {
            if (input[i] > (23 + offset[25]))
            {
                if (input[i] > (27 + offset[25]))
                {
                    if (input[i] > (29 + offset[25]))
                    {
                        if (input[i] == (31 + offset[25]))
                        {
                            vR[i] = 31;
                        }
                        else
                        {
                            vR[i] = 30;
                        }
                    }
                    else
                    {
                        if (input[i] == (29 + offset[25]))
                        {
                            vR[i] = 29;
                        }
                        else
                        {
                            vR[i] = 28;
                        }
                    }
                }
                else
                {
                    if (input[i] > (25 + offset[25]))
                    {
                        if (input[i] == (27 + offset[25]))
                        {
                            vR[i] = 27;
                        }
                        else
                        {
                            vR[i] = 26;
                        }
                    }
                    else
                    {
                        if (input[i] == (25 + offset[25]))
                        {
                            vR[i] = 25;
                        }
                        else
                        {
                            vR[i] = 24;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (19 + offset[25]))
                {
                    if (input[i] > (21 + offset[25]))
                    {
                        if (input[i] == (23 + offset[25]))
                        {
                            vR[i] = 23;
                        }
                        else
                        {
                            vR[i] = 22;
                        }
                    }
                    else
                    {
                        if (input[i] == (21 + offset[25]))
                        {
                            vR[i] = 21;
                        }
                        else
                        {
                            vR[i] = 20;
                        }
                    }
                }
                else
                {
                    if (input[i] > (17 + offset[25]))
                    {
                        if (input[i] == (19 + offset[25]))
                        {
                            vR[i] = 19;
                        }
                        else
                        {
                            vR[i] = 18;
                        }
                    }
                    else
                    {
                        if (input[i] == (17 + offset[25]))
                        {
                            vR[i] = 17;
                        }
                        else
                        {
                            vR[i] = 16;
                        }
                    }
                }
            }
        }
        else
        {
            if (input[i] > (7 + offset[25]))
            {
                if (input[i] > (11 + offset[25]))
                {
                    if (input[i] > (13 + offset[25]))
                    {
                        if (input[i] == (15 + offset[25]))
                        {
                            vR[i] = 15;
                        }
                        else
                        {
                            vR[i] = 14;
                        }
                    }
                    else
                    {
                        if (input[i] == (13 + offset[25]))
                        {
                            vR[i] = 13;
                        }
                        else
                        {
                            vR[i] = 12;
                        }
                    }
                }
                else
                {
                    if (input[i] > (9 + offset[25]))
                    {
                        if (input[i] == (11 + offset[25]))
                        {
                            vR[i] = 11;
                        }
                        else
                        {
                            vR[i] = 10;
                        }
                    }
                    else
                    {
                        if (input[i] == (9 + offset[25]))
                        {
                            vR[i] = 9;
                        }
                        else
                        {
                            vR[i] = 8;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (3 + offset[25]))
                {
                    if (input[i] > (5 + offset[25]))
                    {
                        if (input[i] == (7 + offset[25]))
                        {
                            vR[i] = 7;
                        }
                        else
                        {
                            vR[i] = 6;
                        }
                    }
                    else
                    {
                        if (input[i] == (5 + offset[25]))
                        {
                            vR[i] = 5;
                        }
                        else
                        {
                            vR[i] = 4;
                        }
                    }
                }
                else
                {
                    if (input[i] > (1 + offset[25]))
                    {
                        if (input[i] == (3 + offset[25]))
                        {
                            vR[i] = 3;
                        }
                        else
                        {
                            vR[i] = 2;
                        }
                    }
                    else
                    {
                        if (input[i] == (1 + offset[25]))
                        {
                            vR[i] = 1;
                        }
                        else
                        {
                            vR[i] = 0;
                        }
                    }
                }
            }
        }
    }
    else if (threadIdx.x%WARP_SIZE == 26)
    {
        if (input[i] > (15 + offset[26]))
        {
            if (input[i] > (23 + offset[26]))
            {
                if (input[i] > (27 + offset[26]))
                {
                    if (input[i] > (29 + offset[26]))
                    {
                        if (input[i] == (31 + offset[26]))
                        {
                            vR[i] = 31;
                        }
                        else
                        {
                            vR[i] = 30;
                        }
                    }
                    else
                    {
                        if (input[i] == (29 + offset[26]))
                        {
                            vR[i] = 29;
                        }
                        else
                        {
                            vR[i] = 28;
                        }
                    }
                }
                else
                {
                    if (input[i] > (25 + offset[26]))
                    {
                        if (input[i] == (27 + offset[26]))
                        {
                            vR[i] = 27;
                        }
                        else
                        {
                            vR[i] = 26;
                        }
                    }
                    else
                    {
                        if (input[i] == (25 + offset[26]))
                        {
                            vR[i] = 25;
                        }
                        else
                        {
                            vR[i] = 24;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (19 + offset[26]))
                {
                    if (input[i] > (21 + offset[26]))
                    {
                        if (input[i] == (23 + offset[26]))
                        {
                            vR[i] = 23;
                        }
                        else
                        {
                            vR[i] = 22;
                        }
                    }
                    else
                    {
                        if (input[i] == (21 + offset[26]))
                        {
                            vR[i] = 21;
                        }
                        else
                        {
                            vR[i] = 20;
                        }
                    }
                }
                else
                {
                    if (input[i] > (17 + offset[26]))
                    {
                        if (input[i] == (19 + offset[26]))
                        {
                            vR[i] = 19;
                        }
                        else
                        {
                            vR[i] = 18;
                        }
                    }
                    else
                    {
                        if (input[i] == (17 + offset[26]))
                        {
                            vR[i] = 17;
                        }
                        else
                        {
                            vR[i] = 16;
                        }
                    }
                }
            }
        }
        else
        {
            if (input[i] > (7 + offset[26]))
            {
                if (input[i] > (11 + offset[26]))
                {
                    if (input[i] > (13 + offset[26]))
                    {
                        if (input[i] == (15 + offset[26]))
                        {
                            vR[i] = 15;
                        }
                        else
                        {
                            vR[i] = 14;
                        }
                    }
                    else
                    {
                        if (input[i] == (13 + offset[26]))
                        {
                            vR[i] = 13;
                        }
                        else
                        {
                            vR[i] = 12;
                        }
                    }
                }
                else
                {
                    if (input[i] > (9 + offset[26]))
                    {
                        if (input[i] == (11 + offset[26]))
                        {
                            vR[i] = 11;
                        }
                        else
                        {
                            vR[i] = 10;
                        }
                    }
                    else
                    {
                        if (input[i] == (9 + offset[26]))
                        {
                            vR[i] = 9;
                        }
                        else
                        {
                            vR[i] = 8;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (3 + offset[26]))
                {
                    if (input[i] > (5 + offset[26]))
                    {
                        if (input[i] == (7 + offset[26]))
                        {
                            vR[i] = 7;
                        }
                        else
                        {
                            vR[i] = 6;
                        }
                    }
                    else
                    {
                        if (input[i] == (5 + offset[26]))
                        {
                            vR[i] = 5;
                        }
                        else
                        {
                            vR[i] = 4;
                        }
                    }
                }
                else
                {
                    if (input[i] > (1 + offset[26]))
                    {
                        if (input[i] == (3 + offset[26]))
                        {
                            vR[i] = 3;
                        }
                        else
                        {
                            vR[i] = 2;
                        }
                    }
                    else
                    {
                        if (input[i] == (1 + offset[26]))
                        {
                            vR[i] = 1;
                        }
                        else
                        {
                            vR[i] = 0;
                        }
                    }
                }
            }
        }
    }
    else if (threadIdx.x%WARP_SIZE == 27)
    {
        if (input[i] > (15 + offset[27]))
        {
            if (input[i] > (23 + offset[27]))
            {
                if (input[i] > (27 + offset[27]))
                {
                    if (input[i] > (29 + offset[27]))
                    {
                        if (input[i] == (31 + offset[27]))
                        {
                            vR[i] = 31;
                        }
                        else
                        {
                            vR[i] = 30;
                        }
                    }
                    else
                    {
                        if (input[i] == (29 + offset[27]))
                        {
                            vR[i] = 29;
                        }
                        else
                        {
                            vR[i] = 28;
                        }
                    }
                }
                else
                {
                    if (input[i] > (25 + offset[27]))
                    {
                        if (input[i] == (27 + offset[27]))
                        {
                            vR[i] = 27;
                        }
                        else
                        {
                            vR[i] = 26;
                        }
                    }
                    else
                    {
                        if (input[i] == (25 + offset[27]))
                        {
                            vR[i] = 25;
                        }
                        else
                        {
                            vR[i] = 24;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (19 + offset[27]))
                {
                    if (input[i] > (21 + offset[27]))
                    {
                        if (input[i] == (23 + offset[27]))
                        {
                            vR[i] = 23;
                        }
                        else
                        {
                            vR[i] = 22;
                        }
                    }
                    else
                    {
                        if (input[i] == (21 + offset[27]))
                        {
                            vR[i] = 21;
                        }
                        else
                        {
                            vR[i] = 20;
                        }
                    }
                }
                else
                {
                    if (input[i] > (17 + offset[27]))
                    {
                        if (input[i] == (19 + offset[27]))
                        {
                            vR[i] = 19;
                        }
                        else
                        {
                            vR[i] = 18;
                        }
                    }
                    else
                    {
                        if (input[i] == (17 + offset[27]))
                        {
                            vR[i] = 17;
                        }
                        else
                        {
                            vR[i] = 16;
                        }
                    }
                }
            }
        }
        else
        {
            if (input[i] > (7 + offset[27]))
            {
                if (input[i] > (11 + offset[27]))
                {
                    if (input[i] > (13 + offset[27]))
                    {
                        if (input[i] == (15 + offset[27]))
                        {
                            vR[i] = 15;
                        }
                        else
                        {
                            vR[i] = 14;
                        }
                    }
                    else
                    {
                        if (input[i] == (13 + offset[27]))
                        {
                            vR[i] = 13;
                        }
                        else
                        {
                            vR[i] = 12;
                        }
                    }
                }
                else
                {
                    if (input[i] > (9 + offset[27]))
                    {
                        if (input[i] == (11 + offset[27]))
                        {
                            vR[i] = 11;
                        }
                        else
                        {
                            vR[i] = 10;
                        }
                    }
                    else
                    {
                        if (input[i] == (9 + offset[27]))
                        {
                            vR[i] = 9;
                        }
                        else
                        {
                            vR[i] = 8;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (3 + offset[27]))
                {
                    if (input[i] > (5 + offset[27]))
                    {
                        if (input[i] == (7 + offset[27]))
                        {
                            vR[i] = 7;
                        }
                        else
                        {
                            vR[i] = 6;
                        }
                    }
                    else
                    {
                        if (input[i] == (5 + offset[27]))
                        {
                            vR[i] = 5;
                        }
                        else
                        {
                            vR[i] = 4;
                        }
                    }
                }
                else
                {
                    if (input[i] > (1 + offset[27]))
                    {
                        if (input[i] == (3 + offset[27]))
                        {
                            vR[i] = 3;
                        }
                        else
                        {
                            vR[i] = 2;
                        }
                    }
                    else
                    {
                        if (input[i] == (1 + offset[27]))
                        {
                            vR[i] = 1;
                        }
                        else
                        {
                            vR[i] = 0;
                        }
                    }
                }
            }
        }
    }
    else if (threadIdx.x%WARP_SIZE == 28)
    {
        if (input[i] > (15 + offset[28]))
        {
            if (input[i] > (23 + offset[28]))
            {
                if (input[i] > (27 + offset[28]))
                {
                    if (input[i] > (29 + offset[28]))
                    {
                        if (input[i] == (31 + offset[28]))
                        {
                            vR[i] = 31;
                        }
                        else
                        {
                            vR[i] = 30;
                        }
                    }
                    else
                    {
                        if (input[i] == (29 + offset[28]))
                        {
                            vR[i] = 29;
                        }
                        else
                        {
                            vR[i] = 28;
                        }
                    }
                }
                else
                {
                    if (input[i] > (25 + offset[28]))
                    {
                        if (input[i] == (27 + offset[28]))
                        {
                            vR[i] = 27;
                        }
                        else
                        {
                            vR[i] = 26;
                        }
                    }
                    else
                    {
                        if (input[i] == (25 + offset[28]))
                        {
                            vR[i] = 25;
                        }
                        else
                        {
                            vR[i] = 24;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (19 + offset[28]))
                {
                    if (input[i] > (21 + offset[28]))
                    {
                        if (input[i] == (23 + offset[28]))
                        {
                            vR[i] = 23;
                        }
                        else
                        {
                            vR[i] = 22;
                        }
                    }
                    else
                    {
                        if (input[i] == (21 + offset[28]))
                        {
                            vR[i] = 21;
                        }
                        else
                        {
                            vR[i] = 20;
                        }
                    }
                }
                else
                {
                    if (input[i] > (17 + offset[28]))
                    {
                        if (input[i] == (19 + offset[28]))
                        {
                            vR[i] = 19;
                        }
                        else
                        {
                            vR[i] = 18;
                        }
                    }
                    else
                    {
                        if (input[i] == (17 + offset[28]))
                        {
                            vR[i] = 17;
                        }
                        else
                        {
                            vR[i] = 16;
                        }
                    }
                }
            }
        }
        else
        {
            if (input[i] > (7 + offset[28]))
            {
                if (input[i] > (11 + offset[28]))
                {
                    if (input[i] > (13 + offset[28]))
                    {
                        if (input[i] == (15 + offset[28]))
                        {
                            vR[i] = 15;
                        }
                        else
                        {
                            vR[i] = 14;
                        }
                    }
                    else
                    {
                        if (input[i] == (13 + offset[28]))
                        {
                            vR[i] = 13;
                        }
                        else
                        {
                            vR[i] = 12;
                        }
                    }
                }
                else
                {
                    if (input[i] > (9 + offset[28]))
                    {
                        if (input[i] == (11 + offset[28]))
                        {
                            vR[i] = 11;
                        }
                        else
                        {
                            vR[i] = 10;
                        }
                    }
                    else
                    {
                        if (input[i] == (9 + offset[28]))
                        {
                            vR[i] = 9;
                        }
                        else
                        {
                            vR[i] = 8;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (3 + offset[28]))
                {
                    if (input[i] > (5 + offset[28]))
                    {
                        if (input[i] == (7 + offset[28]))
                        {
                            vR[i] = 7;
                        }
                        else
                        {
                            vR[i] = 6;
                        }
                    }
                    else
                    {
                        if (input[i] == (5 + offset[28]))
                        {
                            vR[i] = 5;
                        }
                        else
                        {
                            vR[i] = 4;
                        }
                    }
                }
                else
                {
                    if (input[i] > (1 + offset[28]))
                    {
                        if (input[i] == (3 + offset[28]))
                        {
                            vR[i] = 3;
                        }
                        else
                        {
                            vR[i] = 2;
                        }
                    }
                    else
                    {
                        if (input[i] == (1 + offset[28]))
                        {
                            vR[i] = 1;
                        }
                        else
                        {
                            vR[i] = 0;
                        }
                    }
                }
            }
        }
    }
    else if (threadIdx.x%WARP_SIZE == 29)
    {
        if (input[i] > (15 + offset[29]))
        {
            if (input[i] > (23 + offset[29]))
            {
                if (input[i] > (27 + offset[29]))
                {
                    if (input[i] > (29 + offset[29]))
                    {
                        if (input[i] == (31 + offset[29]))
                        {
                            vR[i] = 31;
                        }
                        else
                        {
                            vR[i] = 30;
                        }
                    }
                    else
                    {
                        if (input[i] == (29 + offset[29]))
                        {
                            vR[i] = 29;
                        }
                        else
                        {
                            vR[i] = 28;
                        }
                    }
                }
                else
                {
                    if (input[i] > (25 + offset[29]))
                    {
                        if (input[i] == (27 + offset[29]))
                        {
                            vR[i] = 27;
                        }
                        else
                        {
                            vR[i] = 26;
                        }
                    }
                    else
                    {
                        if (input[i] == (25 + offset[29]))
                        {
                            vR[i] = 25;
                        }
                        else
                        {
                            vR[i] = 24;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (19 + offset[29]))
                {
                    if (input[i] > (21 + offset[29]))
                    {
                        if (input[i] == (23 + offset[29]))
                        {
                            vR[i] = 23;
                        }
                        else
                        {
                            vR[i] = 22;
                        }
                    }
                    else
                    {
                        if (input[i] == (21 + offset[29]))
                        {
                            vR[i] = 21;
                        }
                        else
                        {
                            vR[i] = 20;
                        }
                    }
                }
                else
                {
                    if (input[i] > (17 + offset[29]))
                    {
                        if (input[i] == (19 + offset[29]))
                        {
                            vR[i] = 19;
                        }
                        else
                        {
                            vR[i] = 18;
                        }
                    }
                    else
                    {
                        if (input[i] == (17 + offset[29]))
                        {
                            vR[i] = 17;
                        }
                        else
                        {
                            vR[i] = 16;
                        }
                    }
                }
            }
        }
        else
        {
            if (input[i] > (7 + offset[29]))
            {
                if (input[i] > (11 + offset[29]))
                {
                    if (input[i] > (13 + offset[29]))
                    {
                        if (input[i] == (15 + offset[29]))
                        {
                            vR[i] = 15;
                        }
                        else
                        {
                            vR[i] = 14;
                        }
                    }
                    else
                    {
                        if (input[i] == (13 + offset[29]))
                        {
                            vR[i] = 13;
                        }
                        else
                        {
                            vR[i] = 12;
                        }
                    }
                }
                else
                {
                    if (input[i] > (9 + offset[29]))
                    {
                        if (input[i] == (11 + offset[29]))
                        {
                            vR[i] = 11;
                        }
                        else
                        {
                            vR[i] = 10;
                        }
                    }
                    else
                    {
                        if (input[i] == (9 + offset[29]))
                        {
                            vR[i] = 9;
                        }
                        else
                        {
                            vR[i] = 8;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (3 + offset[29]))
                {
                    if (input[i] > (5 + offset[29]))
                    {
                        if (input[i] == (7 + offset[29]))
                        {
                            vR[i] = 7;
                        }
                        else
                        {
                            vR[i] = 6;
                        }
                    }
                    else
                    {
                        if (input[i] == (5 + offset[29]))
                        {
                            vR[i] = 5;
                        }
                        else
                        {
                            vR[i] = 4;
                        }
                    }
                }
                else
                {
                    if (input[i] > (1 + offset[29]))
                    {
                        if (input[i] == (3 + offset[29]))
                        {
                            vR[i] = 3;
                        }
                        else
                        {
                            vR[i] = 2;
                        }
                    }
                    else
                    {
                        if (input[i] == (1 + offset[29]))
                        {
                            vR[i] = 1;
                        }
                        else
                        {
                            vR[i] = 0;
                        }
                    }
                }
            }
        }
    }
    else if (threadIdx.x%WARP_SIZE == 30)
    {
        if (input[i] > (15 + offset[30]))
        {
            if (input[i] > (23 + offset[30]))
            {
                if (input[i] > (27 + offset[30]))
                {
                    if (input[i] > (29 + offset[30]))
                    {
                        if (input[i] == (31 + offset[30]))
                        {
                            vR[i] = 31;
                        }
                        else
                        {
                            vR[i] = 30;
                        }
                    }
                    else
                    {
                        if (input[i] == (29 + offset[30]))
                        {
                            vR[i] = 29;
                        }
                        else
                        {
                            vR[i] = 28;
                        }
                    }
                }
                else
                {
                    if (input[i] > (25 + offset[30]))
                    {
                        if (input[i] == (27 + offset[30]))
                        {
                            vR[i] = 27;
                        }
                        else
                        {
                            vR[i] = 26;
                        }
                    }
                    else
                    {
                        if (input[i] == (25 + offset[30]))
                        {
                            vR[i] = 25;
                        }
                        else
                        {
                            vR[i] = 24;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (19 + offset[30]))
                {
                    if (input[i] > (21 + offset[30]))
                    {
                        if (input[i] == (23 + offset[30]))
                        {
                            vR[i] = 23;
                        }
                        else
                        {
                            vR[i] = 22;
                        }
                    }
                    else
                    {
                        if (input[i] == (21 + offset[30]))
                        {
                            vR[i] = 21;
                        }
                        else
                        {
                            vR[i] = 20;
                        }
                    }
                }
                else
                {
                    if (input[i] > (17 + offset[30]))
                    {
                        if (input[i] == (19 + offset[30]))
                        {
                            vR[i] = 19;
                        }
                        else
                        {
                            vR[i] = 18;
                        }
                    }
                    else
                    {
                        if (input[i] == (17 + offset[30]))
                        {
                            vR[i] = 17;
                        }
                        else
                        {
                            vR[i] = 16;
                        }
                    }
                }
            }
        }
        else
        {
            if (input[i] > (7 + offset[30]))
            {
                if (input[i] > (11 + offset[30]))
                {
                    if (input[i] > (13 + offset[30]))
                    {
                        if (input[i] == (15 + offset[30]))
                        {
                            vR[i] = 15;
                        }
                        else
                        {
                            vR[i] = 14;
                        }
                    }
                    else
                    {
                        if (input[i] == (13 + offset[30]))
                        {
                            vR[i] = 13;
                        }
                        else
                        {
                            vR[i] = 12;
                        }
                    }
                }
                else
                {
                    if (input[i] > (9 + offset[30]))
                    {
                        if (input[i] == (11 + offset[30]))
                        {
                            vR[i] = 11;
                        }
                        else
                        {
                            vR[i] = 10;
                        }
                    }
                    else
                    {
                        if (input[i] == (9 + offset[30]))
                        {
                            vR[i] = 9;
                        }
                        else
                        {
                            vR[i] = 8;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (3 + offset[30]))
                {
                    if (input[i] > (5 + offset[30]))
                    {
                        if (input[i] == (7 + offset[30]))
                        {
                            vR[i] = 7;
                        }
                        else
                        {
                            vR[i] = 6;
                        }
                    }
                    else
                    {
                        if (input[i] == (5 + offset[30]))
                        {
                            vR[i] = 5;
                        }
                        else
                        {
                            vR[i] = 4;
                        }
                    }
                }
                else
                {
                    if (input[i] > (1 + offset[30]))
                    {
                        if (input[i] == (3 + offset[30]))
                        {
                            vR[i] = 3;
                        }
                        else
                        {
                            vR[i] = 2;
                        }
                    }
                    else
                    {
                        if (input[i] == (1 + offset[30]))
                        {
                            vR[i] = 1;
                        }
                        else
                        {
                            vR[i] = 0;
                        }
                    }
                }
            }
        }
    }
    else if (threadIdx.x%WARP_SIZE == 31)
    {
        if (input[i] > (15 + offset[31]))
        {
            if (input[i] > (23 + offset[31]))
            {
                if (input[i] > (27 + offset[31]))
                {
                    if (input[i] > (29 + offset[31]))
                    {
                        if (input[i] == (31 + offset[31]))
                        {
                            vR[i] = 31;
                        }
                        else
                        {
                            vR[i] = 30;
                        }
                    }
                    else
                    {
                        if (input[i] == (29 + offset[31]))
                        {
                            vR[i] = 29;
                        }
                        else
                        {
                            vR[i] = 28;
                        }
                    }
                }
                else
                {
                    if (input[i] > (25 + offset[31]))
                    {
                        if (input[i] == (27 + offset[31]))
                        {
                            vR[i] = 27;
                        }
                        else
                        {
                            vR[i] = 26;
                        }
                    }
                    else
                    {
                        if (input[i] == (25 + offset[31]))
                        {
                            vR[i] = 25;
                        }
                        else
                        {
                            vR[i] = 24;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (19 + offset[31]))
                {
                    if (input[i] > (21 + offset[31]))
                    {
                        if (input[i] == (23 + offset[31]))
                        {
                            vR[i] = 23;
                        }
                        else
                        {
                            vR[i] = 22;
                        }
                    }
                    else
                    {
                        if (input[i] == (21 + offset[31]))
                        {
                            vR[i] = 21;
                        }
                        else
                        {
                            vR[i] = 20;
                        }
                    }
                }
                else
                {
                    if (input[i] > (17 + offset[31]))
                    {
                        if (input[i] == (19 + offset[31]))
                        {
                            vR[i] = 19;
                        }
                        else
                        {
                            vR[i] = 18;
                        }
                    }
                    else
                    {
                        if (input[i] == (17 + offset[31]))
                        {
                            vR[i] = 17;
                        }
                        else
                        {
                            vR[i] = 16;
                        }
                    }
                }
            }
        }
        else
        {
            if (input[i] > (7 + offset[31]))
            {
                if (input[i] > (11 + offset[31]))
                {
                    if (input[i] > (13 + offset[31]))
                    {
                        if (input[i] == (15 + offset[31]))
                        {
                            vR[i] = 15;
                        }
                        else
                        {
                            vR[i] = 14;
                        }
                    }
                    else
                    {
                        if (input[i] == (13 + offset[31]))
                        {
                            vR[i] = 13;
                        }
                        else
                        {
                            vR[i] = 12;
                        }
                    }
                }
                else
                {
                    if (input[i] > (9 + offset[31]))
                    {
                        if (input[i] == (11 + offset[31]))
                        {
                            vR[i] = 11;
                        }
                        else
                        {
                            vR[i] = 10;
                        }
                    }
                    else
                    {
                        if (input[i] == (9 + offset[31]))
                        {
                            vR[i] = 9;
                        }
                        else
                        {
                            vR[i] = 8;
                        }
                    }
                }
            }
            else
            {
                if (input[i] > (3 + offset[31]))
                {
                    if (input[i] > (5 + offset[31]))
                    {
                        if (input[i] == (7 + offset[31]))
                        {
                            vR[i] = 7;
                        }
                        else
                        {
                            vR[i] = 6;
                        }
                    }
                    else
                    {
                        if (input[i] == (5 + offset[31]))
                        {
                            vR[i] = 5;
                        }
                        else
                        {
                            vR[i] = 4;
                        }
                    }
                }
                else
                {
                    if (input[i] > (1 + offset[31]))
                    {
                        if (input[i] == (3 + offset[31]))
                        {
                            vR[i] = 3;
                        }
                        else
                        {
                            vR[i] = 2;
                        }
                    }
                    else
                    {
                        if (input[i] == (1 + offset[31]))
                        {
                            vR[i] = 1;
                        }
                        else
                        {
                            vR[i] = 0;
                        }
                    }
                }
            }
        }
    }
}

void initData(float *ip, int size);
void registerTime(float value);

int main(int argc, char **argv)
{
    float elapsed_time;
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("[CUDA - Random Forest]: EXP %s\n", EXP_NAME);
    printf("[CUDA - Random Forest]: Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set up data size of vectors
    int nElem = N_ELEM;

    printf("[CUDA - Random Forest]: Vector Size %d\n", nElem);

    // malloc host memory
    size_t nBytes = nElem * sizeof(float);
    size_t offsetBytes = WARP_SIZE * sizeof(int);

    float *h_vR;
    h_vR = (float *)malloc(nBytes);
    float *h_input;
    h_input = (float *)malloc(nBytes);
    int *h_offset = (int *)malloc(offsetBytes);

    printf("[CUDA - Random Forest]: Start Reading Dataset\n");
    initData(h_input, nElem);
    printf("[CUDA - Random Forest]: Finished Reading Dataset\n");
    memset(h_vR, 0, nBytes);
    memset(h_offset, 0, offsetBytes);

    // malloc device global memory
    float *d_vR;
    CHECK(hipMalloc((float **)&d_vR, nBytes));
    float *d_input;
    CHECK(hipMalloc((float **)&d_input, nBytes));
    int *d_offset;
    CHECK(hipMalloc((float **)&d_offset, offsetBytes));

    // transfer data from host to device
    CHECK(hipMemcpy(d_input, h_input, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_offset, h_offset, offsetBytes, hipMemcpyHostToDevice));

    // invoke kernel at host side
    int iLen = 512; // 1024
    dim3 block(iLen);
    dim3 grid((nElem + block.x - 1) / block.x); // /2
    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    // record start event
    CHECK(hipEventRecord(start, 0));


    RF_kernel<<<grid, block>>>(d_input, nElem, d_offset, d_vR);

    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    hipDeviceSynchronize();

    // calculate elapsed time
    CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
    printf("[CUDA - Random Forest]: %s - execution time = %.6fms\n", EXP_NAME, elapsed_time);
    registerTime(elapsed_time);

    // copy kernel result back to host side
    CHECK(hipMemcpy(h_vR, d_vR, nBytes, hipMemcpyDeviceToHost));
    CHECK(hipGetLastError());

    // Verify the answers
    char flag = 1;
    for (int i = 0; i < nElem; i++)
    {
        if (i < WARP_SIZE)
        {
            printf("%.3f ", h_vR[i]);
        }
        if (h_input[i] != h_vR[i])
        {
            flag = 0;
            // break;
        }
    }
    if (flag == 1)
    {
        printf("Return correct");
    }
    else
    {
        printf("Error!!");
    }
    printf("\n ");

    // free device memory
    hipFree(d_input);
    hipFree(d_vR);
    hipFree(d_offset);


    // free host memory
    free(h_input);
    free(h_vR);
    free(h_offset);

    return (0);
}

void initData(float *ip, int size)
{
    for (int i = 0; i < size; i++)
    {
        if (WARP_SIZE <= _MOD)
        {
            int aux = _MOD / WARP_SIZE;
            ip[i] = (float)((i * aux) % _MOD);
        }
        else
        {
            ip[i] = (float)(i % _MOD);
        }
        // if (i < 32)
        //	printf("%.2f ", ip[i]);
    }
    return;
}

void registerTime(float value)
{
    printf("[CUDA - Random Forest]: Time spent: %f.\n", value);
}
