#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>

#define _WORST
#define N_ELEM 100000000

#define WARP_SIZE 32

#define CHECK(call)                                                \
	{                                                              \
		const hipError_t error = call;                            \
		if (error != hipSuccess)                                  \
		{                                                          \
			fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
			fprintf(stderr, "code: %d, reason: %s\n", error,       \
					hipGetErrorString(error));                    \
		}                                                          \
	}

void initData(float *ip, int size);
void registerTime(float value);

#define _MOD 128
#define _2TREE
#define EXP_NAME "_RF_127IF_2TREE_1INPUT__WORST_100000000"

// RF_with_if

__global__ void RF_kernel(float *input, const int n, const int offset0, const int offset1, float *vR)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
if(input[i]>(63+offset0))
{
	if(input[i]>(95+offset0))
{
	if(input[i]>(111+offset0))
{
	if(input[i]>(119+offset0))
{
	if(input[i]>(123+offset0))
{
	if(input[i]>(125+offset0))
{
	if(input[i]==(127+offset0))
{
	vR[i] = 127;
}
else
{
	vR[i] = 126;
}

}
else
{
	if(input[i]==(125+offset0))
{
	vR[i] = 125;
}
else
{
	vR[i] = 124;
}

}

}
else
{
	if(input[i]>(121+offset0))
{
	if(input[i]==(123+offset0))
{
	vR[i] = 123;
}
else
{
	vR[i] = 122;
}

}
else
{
	if(input[i]==(121+offset0))
{
	vR[i] = 121;
}
else
{
	vR[i] = 120;
}

}

}

}
else
{
	if(input[i]>(115+offset0))
{
	if(input[i]>(117+offset0))
{
	if(input[i]==(119+offset0))
{
	vR[i] = 119;
}
else
{
	vR[i] = 118;
}

}
else
{
	if(input[i]==(117+offset0))
{
	vR[i] = 117;
}
else
{
	vR[i] = 116;
}

}

}
else
{
	if(input[i]>(113+offset0))
{
	if(input[i]==(115+offset0))
{
	vR[i] = 115;
}
else
{
	vR[i] = 114;
}

}
else
{
	if(input[i]==(113+offset0))
{
	vR[i] = 113;
}
else
{
	vR[i] = 112;
}

}

}

}

}
else
{
	if(input[i]>(103+offset0))
{
	if(input[i]>(107+offset0))
{
	if(input[i]>(109+offset0))
{
	if(input[i]==(111+offset0))
{
	vR[i] = 111;
}
else
{
	vR[i] = 110;
}

}
else
{
	if(input[i]==(109+offset0))
{
	vR[i] = 109;
}
else
{
	vR[i] = 108;
}

}

}
else
{
	if(input[i]>(105+offset0))
{
	if(input[i]==(107+offset0))
{
	vR[i] = 107;
}
else
{
	vR[i] = 106;
}

}
else
{
	if(input[i]==(105+offset0))
{
	vR[i] = 105;
}
else
{
	vR[i] = 104;
}

}

}

}
else
{
	if(input[i]>(99+offset0))
{
	if(input[i]>(101+offset0))
{
	if(input[i]==(103+offset0))
{
	vR[i] = 103;
}
else
{
	vR[i] = 102;
}

}
else
{
	if(input[i]==(101+offset0))
{
	vR[i] = 101;
}
else
{
	vR[i] = 100;
}

}

}
else
{
	if(input[i]>(97+offset0))
{
	if(input[i]==(99+offset0))
{
	vR[i] = 99;
}
else
{
	vR[i] = 98;
}

}
else
{
	if(input[i]==(97+offset0))
{
	vR[i] = 97;
}
else
{
	vR[i] = 96;
}

}

}

}

}

}
else
{
	if(input[i]>(79+offset0))
{
	if(input[i]>(87+offset0))
{
	if(input[i]>(91+offset0))
{
	if(input[i]>(93+offset0))
{
	if(input[i]==(95+offset0))
{
	vR[i] = 95;
}
else
{
	vR[i] = 94;
}

}
else
{
	if(input[i]==(93+offset0))
{
	vR[i] = 93;
}
else
{
	vR[i] = 92;
}

}

}
else
{
	if(input[i]>(89+offset0))
{
	if(input[i]==(91+offset0))
{
	vR[i] = 91;
}
else
{
	vR[i] = 90;
}

}
else
{
	if(input[i]==(89+offset0))
{
	vR[i] = 89;
}
else
{
	vR[i] = 88;
}

}

}

}
else
{
	if(input[i]>(83+offset0))
{
	if(input[i]>(85+offset0))
{
	if(input[i]==(87+offset0))
{
	vR[i] = 87;
}
else
{
	vR[i] = 86;
}

}
else
{
	if(input[i]==(85+offset0))
{
	vR[i] = 85;
}
else
{
	vR[i] = 84;
}

}

}
else
{
	if(input[i]>(81+offset0))
{
	if(input[i]==(83+offset0))
{
	vR[i] = 83;
}
else
{
	vR[i] = 82;
}

}
else
{
	if(input[i]==(81+offset0))
{
	vR[i] = 81;
}
else
{
	vR[i] = 80;
}

}

}

}

}
else
{
	if(input[i]>(71+offset0))
{
	if(input[i]>(75+offset0))
{
	if(input[i]>(77+offset0))
{
	if(input[i]==(79+offset0))
{
	vR[i] = 79;
}
else
{
	vR[i] = 78;
}

}
else
{
	if(input[i]==(77+offset0))
{
	vR[i] = 77;
}
else
{
	vR[i] = 76;
}

}

}
else
{
	if(input[i]>(73+offset0))
{
	if(input[i]==(75+offset0))
{
	vR[i] = 75;
}
else
{
	vR[i] = 74;
}

}
else
{
	if(input[i]==(73+offset0))
{
	vR[i] = 73;
}
else
{
	vR[i] = 72;
}

}

}

}
else
{
	if(input[i]>(67+offset0))
{
	if(input[i]>(69+offset0))
{
	if(input[i]==(71+offset0))
{
	vR[i] = 71;
}
else
{
	vR[i] = 70;
}

}
else
{
	if(input[i]==(69+offset0))
{
	vR[i] = 69;
}
else
{
	vR[i] = 68;
}

}

}
else
{
	if(input[i]>(65+offset0))
{
	if(input[i]==(67+offset0))
{
	vR[i] = 67;
}
else
{
	vR[i] = 66;
}

}
else
{
	if(input[i]==(65+offset0))
{
	vR[i] = 65;
}
else
{
	vR[i] = 64;
}

}

}

}

}

}

}
else
{
	if(input[i]>(31+offset0))
{
	if(input[i]>(47+offset0))
{
	if(input[i]>(55+offset0))
{
	if(input[i]>(59+offset0))
{
	if(input[i]>(61+offset0))
{
	if(input[i]==(63+offset0))
{
	vR[i] = 63;
}
else
{
	vR[i] = 62;
}

}
else
{
	if(input[i]==(61+offset0))
{
	vR[i] = 61;
}
else
{
	vR[i] = 60;
}

}

}
else
{
	if(input[i]>(57+offset0))
{
	if(input[i]==(59+offset0))
{
	vR[i] = 59;
}
else
{
	vR[i] = 58;
}

}
else
{
	if(input[i]==(57+offset0))
{
	vR[i] = 57;
}
else
{
	vR[i] = 56;
}

}

}

}
else
{
	if(input[i]>(51+offset0))
{
	if(input[i]>(53+offset0))
{
	if(input[i]==(55+offset0))
{
	vR[i] = 55;
}
else
{
	vR[i] = 54;
}

}
else
{
	if(input[i]==(53+offset0))
{
	vR[i] = 53;
}
else
{
	vR[i] = 52;
}

}

}
else
{
	if(input[i]>(49+offset0))
{
	if(input[i]==(51+offset0))
{
	vR[i] = 51;
}
else
{
	vR[i] = 50;
}

}
else
{
	if(input[i]==(49+offset0))
{
	vR[i] = 49;
}
else
{
	vR[i] = 48;
}

}

}

}

}
else
{
	if(input[i]>(39+offset0))
{
	if(input[i]>(43+offset0))
{
	if(input[i]>(45+offset0))
{
	if(input[i]==(47+offset0))
{
	vR[i] = 47;
}
else
{
	vR[i] = 46;
}

}
else
{
	if(input[i]==(45+offset0))
{
	vR[i] = 45;
}
else
{
	vR[i] = 44;
}

}

}
else
{
	if(input[i]>(41+offset0))
{
	if(input[i]==(43+offset0))
{
	vR[i] = 43;
}
else
{
	vR[i] = 42;
}

}
else
{
	if(input[i]==(41+offset0))
{
	vR[i] = 41;
}
else
{
	vR[i] = 40;
}

}

}

}
else
{
	if(input[i]>(35+offset0))
{
	if(input[i]>(37+offset0))
{
	if(input[i]==(39+offset0))
{
	vR[i] = 39;
}
else
{
	vR[i] = 38;
}

}
else
{
	if(input[i]==(37+offset0))
{
	vR[i] = 37;
}
else
{
	vR[i] = 36;
}

}

}
else
{
	if(input[i]>(33+offset0))
{
	if(input[i]==(35+offset0))
{
	vR[i] = 35;
}
else
{
	vR[i] = 34;
}

}
else
{
	if(input[i]==(33+offset0))
{
	vR[i] = 33;
}
else
{
	vR[i] = 32;
}

}

}

}

}

}
else
{
	if(input[i]>(15+offset0))
{
	if(input[i]>(23+offset0))
{
	if(input[i]>(27+offset0))
{
	if(input[i]>(29+offset0))
{
	if(input[i]==(31+offset0))
{
	vR[i] = 31;
}
else
{
	vR[i] = 30;
}

}
else
{
	if(input[i]==(29+offset0))
{
	vR[i] = 29;
}
else
{
	vR[i] = 28;
}

}

}
else
{
	if(input[i]>(25+offset0))
{
	if(input[i]==(27+offset0))
{
	vR[i] = 27;
}
else
{
	vR[i] = 26;
}

}
else
{
	if(input[i]==(25+offset0))
{
	vR[i] = 25;
}
else
{
	vR[i] = 24;
}

}

}

}
else
{
	if(input[i]>(19+offset0))
{
	if(input[i]>(21+offset0))
{
	if(input[i]==(23+offset0))
{
	vR[i] = 23;
}
else
{
	vR[i] = 22;
}

}
else
{
	if(input[i]==(21+offset0))
{
	vR[i] = 21;
}
else
{
	vR[i] = 20;
}

}

}
else
{
	if(input[i]>(17+offset0))
{
	if(input[i]==(19+offset0))
{
	vR[i] = 19;
}
else
{
	vR[i] = 18;
}

}
else
{
	if(input[i]==(17+offset0))
{
	vR[i] = 17;
}
else
{
	vR[i] = 16;
}

}

}

}

}
else
{
	if(input[i]>(7+offset0))
{
	if(input[i]>(11+offset0))
{
	if(input[i]>(13+offset0))
{
	if(input[i]==(15+offset0))
{
	vR[i] = 15;
}
else
{
	vR[i] = 14;
}

}
else
{
	if(input[i]==(13+offset0))
{
	vR[i] = 13;
}
else
{
	vR[i] = 12;
}

}

}
else
{
	if(input[i]>(9+offset0))
{
	if(input[i]==(11+offset0))
{
	vR[i] = 11;
}
else
{
	vR[i] = 10;
}

}
else
{
	if(input[i]==(9+offset0))
{
	vR[i] = 9;
}
else
{
	vR[i] = 8;
}

}

}

}
else
{
	if(input[i]>(3+offset0))
{
	if(input[i]>(5+offset0))
{
	if(input[i]==(7+offset0))
{
	vR[i] = 7;
}
else
{
	vR[i] = 6;
}

}
else
{
	if(input[i]==(5+offset0))
{
	vR[i] = 5;
}
else
{
	vR[i] = 4;
}

}

}
else
{
	if(input[i]>(1+offset0))
{
	if(input[i]==(3+offset0))
{
	vR[i] = 3;
}
else
{
	vR[i] = 2;
}

}
else
{
	if(input[i]==(1+offset0))
{
	vR[i] = 1;
}
else
{
	vR[i] = 0;
}

}

}

}

}

}

}


if(input[i]>(63+offset1))
{
	if(input[i]>(95+offset1))
{
	if(input[i]>(111+offset1))
{
	if(input[i]>(119+offset1))
{
	if(input[i]>(123+offset1))
{
	if(input[i]>(125+offset1))
{
	if(input[i]==(127+offset1))
{
	vR[i] = 127;
}
else
{
	vR[i] = 126;
}

}
else
{
	if(input[i]==(125+offset1))
{
	vR[i] = 125;
}
else
{
	vR[i] = 124;
}

}

}
else
{
	if(input[i]>(121+offset1))
{
	if(input[i]==(123+offset1))
{
	vR[i] = 123;
}
else
{
	vR[i] = 122;
}

}
else
{
	if(input[i]==(121+offset1))
{
	vR[i] = 121;
}
else
{
	vR[i] = 120;
}

}

}

}
else
{
	if(input[i]>(115+offset1))
{
	if(input[i]>(117+offset1))
{
	if(input[i]==(119+offset1))
{
	vR[i] = 119;
}
else
{
	vR[i] = 118;
}

}
else
{
	if(input[i]==(117+offset1))
{
	vR[i] = 117;
}
else
{
	vR[i] = 116;
}

}

}
else
{
	if(input[i]>(113+offset1))
{
	if(input[i]==(115+offset1))
{
	vR[i] = 115;
}
else
{
	vR[i] = 114;
}

}
else
{
	if(input[i]==(113+offset1))
{
	vR[i] = 113;
}
else
{
	vR[i] = 112;
}

}

}

}

}
else
{
	if(input[i]>(103+offset1))
{
	if(input[i]>(107+offset1))
{
	if(input[i]>(109+offset1))
{
	if(input[i]==(111+offset1))
{
	vR[i] = 111;
}
else
{
	vR[i] = 110;
}

}
else
{
	if(input[i]==(109+offset1))
{
	vR[i] = 109;
}
else
{
	vR[i] = 108;
}

}

}
else
{
	if(input[i]>(105+offset1))
{
	if(input[i]==(107+offset1))
{
	vR[i] = 107;
}
else
{
	vR[i] = 106;
}

}
else
{
	if(input[i]==(105+offset1))
{
	vR[i] = 105;
}
else
{
	vR[i] = 104;
}

}

}

}
else
{
	if(input[i]>(99+offset1))
{
	if(input[i]>(101+offset1))
{
	if(input[i]==(103+offset1))
{
	vR[i] = 103;
}
else
{
	vR[i] = 102;
}

}
else
{
	if(input[i]==(101+offset1))
{
	vR[i] = 101;
}
else
{
	vR[i] = 100;
}

}

}
else
{
	if(input[i]>(97+offset1))
{
	if(input[i]==(99+offset1))
{
	vR[i] = 99;
}
else
{
	vR[i] = 98;
}

}
else
{
	if(input[i]==(97+offset1))
{
	vR[i] = 97;
}
else
{
	vR[i] = 96;
}

}

}

}

}

}
else
{
	if(input[i]>(79+offset1))
{
	if(input[i]>(87+offset1))
{
	if(input[i]>(91+offset1))
{
	if(input[i]>(93+offset1))
{
	if(input[i]==(95+offset1))
{
	vR[i] = 95;
}
else
{
	vR[i] = 94;
}

}
else
{
	if(input[i]==(93+offset1))
{
	vR[i] = 93;
}
else
{
	vR[i] = 92;
}

}

}
else
{
	if(input[i]>(89+offset1))
{
	if(input[i]==(91+offset1))
{
	vR[i] = 91;
}
else
{
	vR[i] = 90;
}

}
else
{
	if(input[i]==(89+offset1))
{
	vR[i] = 89;
}
else
{
	vR[i] = 88;
}

}

}

}
else
{
	if(input[i]>(83+offset1))
{
	if(input[i]>(85+offset1))
{
	if(input[i]==(87+offset1))
{
	vR[i] = 87;
}
else
{
	vR[i] = 86;
}

}
else
{
	if(input[i]==(85+offset1))
{
	vR[i] = 85;
}
else
{
	vR[i] = 84;
}

}

}
else
{
	if(input[i]>(81+offset1))
{
	if(input[i]==(83+offset1))
{
	vR[i] = 83;
}
else
{
	vR[i] = 82;
}

}
else
{
	if(input[i]==(81+offset1))
{
	vR[i] = 81;
}
else
{
	vR[i] = 80;
}

}

}

}

}
else
{
	if(input[i]>(71+offset1))
{
	if(input[i]>(75+offset1))
{
	if(input[i]>(77+offset1))
{
	if(input[i]==(79+offset1))
{
	vR[i] = 79;
}
else
{
	vR[i] = 78;
}

}
else
{
	if(input[i]==(77+offset1))
{
	vR[i] = 77;
}
else
{
	vR[i] = 76;
}

}

}
else
{
	if(input[i]>(73+offset1))
{
	if(input[i]==(75+offset1))
{
	vR[i] = 75;
}
else
{
	vR[i] = 74;
}

}
else
{
	if(input[i]==(73+offset1))
{
	vR[i] = 73;
}
else
{
	vR[i] = 72;
}

}

}

}
else
{
	if(input[i]>(67+offset1))
{
	if(input[i]>(69+offset1))
{
	if(input[i]==(71+offset1))
{
	vR[i] = 71;
}
else
{
	vR[i] = 70;
}

}
else
{
	if(input[i]==(69+offset1))
{
	vR[i] = 69;
}
else
{
	vR[i] = 68;
}

}

}
else
{
	if(input[i]>(65+offset1))
{
	if(input[i]==(67+offset1))
{
	vR[i] = 67;
}
else
{
	vR[i] = 66;
}

}
else
{
	if(input[i]==(65+offset1))
{
	vR[i] = 65;
}
else
{
	vR[i] = 64;
}

}

}

}

}

}

}
else
{
	if(input[i]>(31+offset1))
{
	if(input[i]>(47+offset1))
{
	if(input[i]>(55+offset1))
{
	if(input[i]>(59+offset1))
{
	if(input[i]>(61+offset1))
{
	if(input[i]==(63+offset1))
{
	vR[i] = 63;
}
else
{
	vR[i] = 62;
}

}
else
{
	if(input[i]==(61+offset1))
{
	vR[i] = 61;
}
else
{
	vR[i] = 60;
}

}

}
else
{
	if(input[i]>(57+offset1))
{
	if(input[i]==(59+offset1))
{
	vR[i] = 59;
}
else
{
	vR[i] = 58;
}

}
else
{
	if(input[i]==(57+offset1))
{
	vR[i] = 57;
}
else
{
	vR[i] = 56;
}

}

}

}
else
{
	if(input[i]>(51+offset1))
{
	if(input[i]>(53+offset1))
{
	if(input[i]==(55+offset1))
{
	vR[i] = 55;
}
else
{
	vR[i] = 54;
}

}
else
{
	if(input[i]==(53+offset1))
{
	vR[i] = 53;
}
else
{
	vR[i] = 52;
}

}

}
else
{
	if(input[i]>(49+offset1))
{
	if(input[i]==(51+offset1))
{
	vR[i] = 51;
}
else
{
	vR[i] = 50;
}

}
else
{
	if(input[i]==(49+offset1))
{
	vR[i] = 49;
}
else
{
	vR[i] = 48;
}

}

}

}

}
else
{
	if(input[i]>(39+offset1))
{
	if(input[i]>(43+offset1))
{
	if(input[i]>(45+offset1))
{
	if(input[i]==(47+offset1))
{
	vR[i] = 47;
}
else
{
	vR[i] = 46;
}

}
else
{
	if(input[i]==(45+offset1))
{
	vR[i] = 45;
}
else
{
	vR[i] = 44;
}

}

}
else
{
	if(input[i]>(41+offset1))
{
	if(input[i]==(43+offset1))
{
	vR[i] = 43;
}
else
{
	vR[i] = 42;
}

}
else
{
	if(input[i]==(41+offset1))
{
	vR[i] = 41;
}
else
{
	vR[i] = 40;
}

}

}

}
else
{
	if(input[i]>(35+offset1))
{
	if(input[i]>(37+offset1))
{
	if(input[i]==(39+offset1))
{
	vR[i] = 39;
}
else
{
	vR[i] = 38;
}

}
else
{
	if(input[i]==(37+offset1))
{
	vR[i] = 37;
}
else
{
	vR[i] = 36;
}

}

}
else
{
	if(input[i]>(33+offset1))
{
	if(input[i]==(35+offset1))
{
	vR[i] = 35;
}
else
{
	vR[i] = 34;
}

}
else
{
	if(input[i]==(33+offset1))
{
	vR[i] = 33;
}
else
{
	vR[i] = 32;
}

}

}

}

}

}
else
{
	if(input[i]>(15+offset1))
{
	if(input[i]>(23+offset1))
{
	if(input[i]>(27+offset1))
{
	if(input[i]>(29+offset1))
{
	if(input[i]==(31+offset1))
{
	vR[i] = 31;
}
else
{
	vR[i] = 30;
}

}
else
{
	if(input[i]==(29+offset1))
{
	vR[i] = 29;
}
else
{
	vR[i] = 28;
}

}

}
else
{
	if(input[i]>(25+offset1))
{
	if(input[i]==(27+offset1))
{
	vR[i] = 27;
}
else
{
	vR[i] = 26;
}

}
else
{
	if(input[i]==(25+offset1))
{
	vR[i] = 25;
}
else
{
	vR[i] = 24;
}

}

}

}
else
{
	if(input[i]>(19+offset1))
{
	if(input[i]>(21+offset1))
{
	if(input[i]==(23+offset1))
{
	vR[i] = 23;
}
else
{
	vR[i] = 22;
}

}
else
{
	if(input[i]==(21+offset1))
{
	vR[i] = 21;
}
else
{
	vR[i] = 20;
}

}

}
else
{
	if(input[i]>(17+offset1))
{
	if(input[i]==(19+offset1))
{
	vR[i] = 19;
}
else
{
	vR[i] = 18;
}

}
else
{
	if(input[i]==(17+offset1))
{
	vR[i] = 17;
}
else
{
	vR[i] = 16;
}

}

}

}

}
else
{
	if(input[i]>(7+offset1))
{
	if(input[i]>(11+offset1))
{
	if(input[i]>(13+offset1))
{
	if(input[i]==(15+offset1))
{
	vR[i] = 15;
}
else
{
	vR[i] = 14;
}

}
else
{
	if(input[i]==(13+offset1))
{
	vR[i] = 13;
}
else
{
	vR[i] = 12;
}

}

}
else
{
	if(input[i]>(9+offset1))
{
	if(input[i]==(11+offset1))
{
	vR[i] = 11;
}
else
{
	vR[i] = 10;
}

}
else
{
	if(input[i]==(9+offset1))
{
	vR[i] = 9;
}
else
{
	vR[i] = 8;
}

}

}

}
else
{
	if(input[i]>(3+offset1))
{
	if(input[i]>(5+offset1))
{
	if(input[i]==(7+offset1))
{
	vR[i] = 7;
}
else
{
	vR[i] = 6;
}

}
else
{
	if(input[i]==(5+offset1))
{
	vR[i] = 5;
}
else
{
	vR[i] = 4;
}

}

}
else
{
	if(input[i]>(1+offset1))
{
	if(input[i]==(3+offset1))
{
	vR[i] = 3;
}
else
{
	vR[i] = 2;
}

}
else
{
	if(input[i]==(1+offset1))
{
	vR[i] = 1;
}
else
{
	vR[i] = 0;
}

}

}

}

}

}

}


	}
}

int main(int argc, char **argv)
{
	float elapsed_time;
	// set up device
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("[CUDA - Random Forest]: EXP %s\n", EXP_NAME);
	printf("[CUDA - Random Forest]: Using Device %d: %s\n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));

	// set up data size of vectors
	int nElem = N_ELEM;
	printf("[CUDA - Random Forest]: Vector Size %d\n", nElem);

	// malloc host memory
	size_t nBytes = nElem * sizeof(float);
	float *h_vR;
	h_vR = (float *)malloc(nBytes);
	float *h_input;
	h_input = (float *)malloc(nBytes);

	printf("[CUDA - Random Forest]: Start Reading Dataset\n");
	initData(h_input, nElem);
	printf("[CUDA - Random Forest]: Finished Reading Dataset\n");
	memset(h_vR, 0, nBytes);

	// malloc device global memory
	float *d_vR;
	CHECK(hipMalloc((float **)&d_vR, nBytes));
	float *d_input;
	CHECK(hipMalloc((float **)&d_input, nBytes));

	// transfer data from host to device
	CHECK(hipMemcpy(d_input, h_input, nBytes, hipMemcpyHostToDevice));

	// invoke kernel at host side
	int iLen = 512;
	dim3 block(iLen);
	dim3 grid((nElem + block.x - 1) / block.x);
	hipEvent_t start, stop;
	CHECK(hipEventCreate(&start));
	CHECK(hipEventCreate(&stop));

	// record start event
	CHECK(hipEventRecord(start, 0));
#if defined(_2TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, d_vR);
#elif defined(_3TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, d_vR);
#elif defined(_4TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, d_vR);
#else
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, d_vR);
#endif
	CHECK(hipEventRecord(stop, 0));
	CHECK(hipEventSynchronize(stop));
	hipDeviceSynchronize();

	// calculate elapsed time
	CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
	printf("[CUDA - Random Forest]: %s - execution time = %.6fms\n", EXP_NAME, elapsed_time);
	registerTime(elapsed_time);

	// copy kernel result back to host side
	CHECK(hipMemcpy(h_vR, d_vR, nBytes, hipMemcpyDeviceToHost));
	CHECK(hipGetLastError());

	//Verify the answers
	char flag = 1;
	for (int i = 0; i < nElem; i++)
	{
		if (i < 32)
		{
			printf("%.3f ", h_vR[i]);
		}
		if (h_input[i] != h_vR[i])
		{
			flag = 0;
			break;
		}
	}
	if (flag == 1)
	{
		printf("Return correct");
	}
	else
	{
		printf("Error!!");
	}
	printf("\n ");

	// free device memory
	hipFree(d_input);
	hipFree(d_vR);

	// free host memory
	free(h_input);
	free(h_vR);
	return (0);
}

void initData(float *ip, int size)
{
	for (int i = 0; i < size; i++)
	{

#if defined(_BEST)
		ip[i] = (float)(0);
#else
		if(WARP_SIZE < _MOD)
		{
			int aux = _MOD / WARP_SIZE;
			ip[i] = (float)((i * aux) % _MOD);
		}
		else
		{
			ip[i] = (float)(i % _MOD);
		}
#endif
	}
	return;
}

void registerTime(float value)
{
	printf("[CUDA - Random Forest]: Time spent: %f.\n", value);
}
