#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>

#define _WORST
#define N_ELEM 100000000

#define WARP_SIZE 32

#define CHECK(call)                                                \
	{                                                              \
		const hipError_t error = call;                            \
		if (error != hipSuccess)                                  \
		{                                                          \
			fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
			fprintf(stderr, "code: %d, reason: %s\n", error,       \
					hipGetErrorString(error));                    \
		}                                                          \
	}

void initData(float *ip, int size);
void registerTime(float value);

#define _MOD 256
#define _1TREE
#define EXP_NAME "_RF_255IF_1TREE_1INPUT__WORST_100000000"

// RF_with_if

__global__ void RF_kernel(float *input, const int n, const int offset0, float *vR)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
if(input[i]>(127+offset0))
{
	if(input[i]>(191+offset0))
{
	if(input[i]>(223+offset0))
{
	if(input[i]>(239+offset0))
{
	if(input[i]>(247+offset0))
{
	if(input[i]>(251+offset0))
{
	if(input[i]>(253+offset0))
{
	if(input[i]==(255+offset0))
{
	vR[i] = 255;
}
else
{
	vR[i] = 254;
}

}
else
{
	if(input[i]==(253+offset0))
{
	vR[i] = 253;
}
else
{
	vR[i] = 252;
}

}

}
else
{
	if(input[i]>(249+offset0))
{
	if(input[i]==(251+offset0))
{
	vR[i] = 251;
}
else
{
	vR[i] = 250;
}

}
else
{
	if(input[i]==(249+offset0))
{
	vR[i] = 249;
}
else
{
	vR[i] = 248;
}

}

}

}
else
{
	if(input[i]>(243+offset0))
{
	if(input[i]>(245+offset0))
{
	if(input[i]==(247+offset0))
{
	vR[i] = 247;
}
else
{
	vR[i] = 246;
}

}
else
{
	if(input[i]==(245+offset0))
{
	vR[i] = 245;
}
else
{
	vR[i] = 244;
}

}

}
else
{
	if(input[i]>(241+offset0))
{
	if(input[i]==(243+offset0))
{
	vR[i] = 243;
}
else
{
	vR[i] = 242;
}

}
else
{
	if(input[i]==(241+offset0))
{
	vR[i] = 241;
}
else
{
	vR[i] = 240;
}

}

}

}

}
else
{
	if(input[i]>(231+offset0))
{
	if(input[i]>(235+offset0))
{
	if(input[i]>(237+offset0))
{
	if(input[i]==(239+offset0))
{
	vR[i] = 239;
}
else
{
	vR[i] = 238;
}

}
else
{
	if(input[i]==(237+offset0))
{
	vR[i] = 237;
}
else
{
	vR[i] = 236;
}

}

}
else
{
	if(input[i]>(233+offset0))
{
	if(input[i]==(235+offset0))
{
	vR[i] = 235;
}
else
{
	vR[i] = 234;
}

}
else
{
	if(input[i]==(233+offset0))
{
	vR[i] = 233;
}
else
{
	vR[i] = 232;
}

}

}

}
else
{
	if(input[i]>(227+offset0))
{
	if(input[i]>(229+offset0))
{
	if(input[i]==(231+offset0))
{
	vR[i] = 231;
}
else
{
	vR[i] = 230;
}

}
else
{
	if(input[i]==(229+offset0))
{
	vR[i] = 229;
}
else
{
	vR[i] = 228;
}

}

}
else
{
	if(input[i]>(225+offset0))
{
	if(input[i]==(227+offset0))
{
	vR[i] = 227;
}
else
{
	vR[i] = 226;
}

}
else
{
	if(input[i]==(225+offset0))
{
	vR[i] = 225;
}
else
{
	vR[i] = 224;
}

}

}

}

}

}
else
{
	if(input[i]>(207+offset0))
{
	if(input[i]>(215+offset0))
{
	if(input[i]>(219+offset0))
{
	if(input[i]>(221+offset0))
{
	if(input[i]==(223+offset0))
{
	vR[i] = 223;
}
else
{
	vR[i] = 222;
}

}
else
{
	if(input[i]==(221+offset0))
{
	vR[i] = 221;
}
else
{
	vR[i] = 220;
}

}

}
else
{
	if(input[i]>(217+offset0))
{
	if(input[i]==(219+offset0))
{
	vR[i] = 219;
}
else
{
	vR[i] = 218;
}

}
else
{
	if(input[i]==(217+offset0))
{
	vR[i] = 217;
}
else
{
	vR[i] = 216;
}

}

}

}
else
{
	if(input[i]>(211+offset0))
{
	if(input[i]>(213+offset0))
{
	if(input[i]==(215+offset0))
{
	vR[i] = 215;
}
else
{
	vR[i] = 214;
}

}
else
{
	if(input[i]==(213+offset0))
{
	vR[i] = 213;
}
else
{
	vR[i] = 212;
}

}

}
else
{
	if(input[i]>(209+offset0))
{
	if(input[i]==(211+offset0))
{
	vR[i] = 211;
}
else
{
	vR[i] = 210;
}

}
else
{
	if(input[i]==(209+offset0))
{
	vR[i] = 209;
}
else
{
	vR[i] = 208;
}

}

}

}

}
else
{
	if(input[i]>(199+offset0))
{
	if(input[i]>(203+offset0))
{
	if(input[i]>(205+offset0))
{
	if(input[i]==(207+offset0))
{
	vR[i] = 207;
}
else
{
	vR[i] = 206;
}

}
else
{
	if(input[i]==(205+offset0))
{
	vR[i] = 205;
}
else
{
	vR[i] = 204;
}

}

}
else
{
	if(input[i]>(201+offset0))
{
	if(input[i]==(203+offset0))
{
	vR[i] = 203;
}
else
{
	vR[i] = 202;
}

}
else
{
	if(input[i]==(201+offset0))
{
	vR[i] = 201;
}
else
{
	vR[i] = 200;
}

}

}

}
else
{
	if(input[i]>(195+offset0))
{
	if(input[i]>(197+offset0))
{
	if(input[i]==(199+offset0))
{
	vR[i] = 199;
}
else
{
	vR[i] = 198;
}

}
else
{
	if(input[i]==(197+offset0))
{
	vR[i] = 197;
}
else
{
	vR[i] = 196;
}

}

}
else
{
	if(input[i]>(193+offset0))
{
	if(input[i]==(195+offset0))
{
	vR[i] = 195;
}
else
{
	vR[i] = 194;
}

}
else
{
	if(input[i]==(193+offset0))
{
	vR[i] = 193;
}
else
{
	vR[i] = 192;
}

}

}

}

}

}

}
else
{
	if(input[i]>(159+offset0))
{
	if(input[i]>(175+offset0))
{
	if(input[i]>(183+offset0))
{
	if(input[i]>(187+offset0))
{
	if(input[i]>(189+offset0))
{
	if(input[i]==(191+offset0))
{
	vR[i] = 191;
}
else
{
	vR[i] = 190;
}

}
else
{
	if(input[i]==(189+offset0))
{
	vR[i] = 189;
}
else
{
	vR[i] = 188;
}

}

}
else
{
	if(input[i]>(185+offset0))
{
	if(input[i]==(187+offset0))
{
	vR[i] = 187;
}
else
{
	vR[i] = 186;
}

}
else
{
	if(input[i]==(185+offset0))
{
	vR[i] = 185;
}
else
{
	vR[i] = 184;
}

}

}

}
else
{
	if(input[i]>(179+offset0))
{
	if(input[i]>(181+offset0))
{
	if(input[i]==(183+offset0))
{
	vR[i] = 183;
}
else
{
	vR[i] = 182;
}

}
else
{
	if(input[i]==(181+offset0))
{
	vR[i] = 181;
}
else
{
	vR[i] = 180;
}

}

}
else
{
	if(input[i]>(177+offset0))
{
	if(input[i]==(179+offset0))
{
	vR[i] = 179;
}
else
{
	vR[i] = 178;
}

}
else
{
	if(input[i]==(177+offset0))
{
	vR[i] = 177;
}
else
{
	vR[i] = 176;
}

}

}

}

}
else
{
	if(input[i]>(167+offset0))
{
	if(input[i]>(171+offset0))
{
	if(input[i]>(173+offset0))
{
	if(input[i]==(175+offset0))
{
	vR[i] = 175;
}
else
{
	vR[i] = 174;
}

}
else
{
	if(input[i]==(173+offset0))
{
	vR[i] = 173;
}
else
{
	vR[i] = 172;
}

}

}
else
{
	if(input[i]>(169+offset0))
{
	if(input[i]==(171+offset0))
{
	vR[i] = 171;
}
else
{
	vR[i] = 170;
}

}
else
{
	if(input[i]==(169+offset0))
{
	vR[i] = 169;
}
else
{
	vR[i] = 168;
}

}

}

}
else
{
	if(input[i]>(163+offset0))
{
	if(input[i]>(165+offset0))
{
	if(input[i]==(167+offset0))
{
	vR[i] = 167;
}
else
{
	vR[i] = 166;
}

}
else
{
	if(input[i]==(165+offset0))
{
	vR[i] = 165;
}
else
{
	vR[i] = 164;
}

}

}
else
{
	if(input[i]>(161+offset0))
{
	if(input[i]==(163+offset0))
{
	vR[i] = 163;
}
else
{
	vR[i] = 162;
}

}
else
{
	if(input[i]==(161+offset0))
{
	vR[i] = 161;
}
else
{
	vR[i] = 160;
}

}

}

}

}

}
else
{
	if(input[i]>(143+offset0))
{
	if(input[i]>(151+offset0))
{
	if(input[i]>(155+offset0))
{
	if(input[i]>(157+offset0))
{
	if(input[i]==(159+offset0))
{
	vR[i] = 159;
}
else
{
	vR[i] = 158;
}

}
else
{
	if(input[i]==(157+offset0))
{
	vR[i] = 157;
}
else
{
	vR[i] = 156;
}

}

}
else
{
	if(input[i]>(153+offset0))
{
	if(input[i]==(155+offset0))
{
	vR[i] = 155;
}
else
{
	vR[i] = 154;
}

}
else
{
	if(input[i]==(153+offset0))
{
	vR[i] = 153;
}
else
{
	vR[i] = 152;
}

}

}

}
else
{
	if(input[i]>(147+offset0))
{
	if(input[i]>(149+offset0))
{
	if(input[i]==(151+offset0))
{
	vR[i] = 151;
}
else
{
	vR[i] = 150;
}

}
else
{
	if(input[i]==(149+offset0))
{
	vR[i] = 149;
}
else
{
	vR[i] = 148;
}

}

}
else
{
	if(input[i]>(145+offset0))
{
	if(input[i]==(147+offset0))
{
	vR[i] = 147;
}
else
{
	vR[i] = 146;
}

}
else
{
	if(input[i]==(145+offset0))
{
	vR[i] = 145;
}
else
{
	vR[i] = 144;
}

}

}

}

}
else
{
	if(input[i]>(135+offset0))
{
	if(input[i]>(139+offset0))
{
	if(input[i]>(141+offset0))
{
	if(input[i]==(143+offset0))
{
	vR[i] = 143;
}
else
{
	vR[i] = 142;
}

}
else
{
	if(input[i]==(141+offset0))
{
	vR[i] = 141;
}
else
{
	vR[i] = 140;
}

}

}
else
{
	if(input[i]>(137+offset0))
{
	if(input[i]==(139+offset0))
{
	vR[i] = 139;
}
else
{
	vR[i] = 138;
}

}
else
{
	if(input[i]==(137+offset0))
{
	vR[i] = 137;
}
else
{
	vR[i] = 136;
}

}

}

}
else
{
	if(input[i]>(131+offset0))
{
	if(input[i]>(133+offset0))
{
	if(input[i]==(135+offset0))
{
	vR[i] = 135;
}
else
{
	vR[i] = 134;
}

}
else
{
	if(input[i]==(133+offset0))
{
	vR[i] = 133;
}
else
{
	vR[i] = 132;
}

}

}
else
{
	if(input[i]>(129+offset0))
{
	if(input[i]==(131+offset0))
{
	vR[i] = 131;
}
else
{
	vR[i] = 130;
}

}
else
{
	if(input[i]==(129+offset0))
{
	vR[i] = 129;
}
else
{
	vR[i] = 128;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(63+offset0))
{
	if(input[i]>(95+offset0))
{
	if(input[i]>(111+offset0))
{
	if(input[i]>(119+offset0))
{
	if(input[i]>(123+offset0))
{
	if(input[i]>(125+offset0))
{
	if(input[i]==(127+offset0))
{
	vR[i] = 127;
}
else
{
	vR[i] = 126;
}

}
else
{
	if(input[i]==(125+offset0))
{
	vR[i] = 125;
}
else
{
	vR[i] = 124;
}

}

}
else
{
	if(input[i]>(121+offset0))
{
	if(input[i]==(123+offset0))
{
	vR[i] = 123;
}
else
{
	vR[i] = 122;
}

}
else
{
	if(input[i]==(121+offset0))
{
	vR[i] = 121;
}
else
{
	vR[i] = 120;
}

}

}

}
else
{
	if(input[i]>(115+offset0))
{
	if(input[i]>(117+offset0))
{
	if(input[i]==(119+offset0))
{
	vR[i] = 119;
}
else
{
	vR[i] = 118;
}

}
else
{
	if(input[i]==(117+offset0))
{
	vR[i] = 117;
}
else
{
	vR[i] = 116;
}

}

}
else
{
	if(input[i]>(113+offset0))
{
	if(input[i]==(115+offset0))
{
	vR[i] = 115;
}
else
{
	vR[i] = 114;
}

}
else
{
	if(input[i]==(113+offset0))
{
	vR[i] = 113;
}
else
{
	vR[i] = 112;
}

}

}

}

}
else
{
	if(input[i]>(103+offset0))
{
	if(input[i]>(107+offset0))
{
	if(input[i]>(109+offset0))
{
	if(input[i]==(111+offset0))
{
	vR[i] = 111;
}
else
{
	vR[i] = 110;
}

}
else
{
	if(input[i]==(109+offset0))
{
	vR[i] = 109;
}
else
{
	vR[i] = 108;
}

}

}
else
{
	if(input[i]>(105+offset0))
{
	if(input[i]==(107+offset0))
{
	vR[i] = 107;
}
else
{
	vR[i] = 106;
}

}
else
{
	if(input[i]==(105+offset0))
{
	vR[i] = 105;
}
else
{
	vR[i] = 104;
}

}

}

}
else
{
	if(input[i]>(99+offset0))
{
	if(input[i]>(101+offset0))
{
	if(input[i]==(103+offset0))
{
	vR[i] = 103;
}
else
{
	vR[i] = 102;
}

}
else
{
	if(input[i]==(101+offset0))
{
	vR[i] = 101;
}
else
{
	vR[i] = 100;
}

}

}
else
{
	if(input[i]>(97+offset0))
{
	if(input[i]==(99+offset0))
{
	vR[i] = 99;
}
else
{
	vR[i] = 98;
}

}
else
{
	if(input[i]==(97+offset0))
{
	vR[i] = 97;
}
else
{
	vR[i] = 96;
}

}

}

}

}

}
else
{
	if(input[i]>(79+offset0))
{
	if(input[i]>(87+offset0))
{
	if(input[i]>(91+offset0))
{
	if(input[i]>(93+offset0))
{
	if(input[i]==(95+offset0))
{
	vR[i] = 95;
}
else
{
	vR[i] = 94;
}

}
else
{
	if(input[i]==(93+offset0))
{
	vR[i] = 93;
}
else
{
	vR[i] = 92;
}

}

}
else
{
	if(input[i]>(89+offset0))
{
	if(input[i]==(91+offset0))
{
	vR[i] = 91;
}
else
{
	vR[i] = 90;
}

}
else
{
	if(input[i]==(89+offset0))
{
	vR[i] = 89;
}
else
{
	vR[i] = 88;
}

}

}

}
else
{
	if(input[i]>(83+offset0))
{
	if(input[i]>(85+offset0))
{
	if(input[i]==(87+offset0))
{
	vR[i] = 87;
}
else
{
	vR[i] = 86;
}

}
else
{
	if(input[i]==(85+offset0))
{
	vR[i] = 85;
}
else
{
	vR[i] = 84;
}

}

}
else
{
	if(input[i]>(81+offset0))
{
	if(input[i]==(83+offset0))
{
	vR[i] = 83;
}
else
{
	vR[i] = 82;
}

}
else
{
	if(input[i]==(81+offset0))
{
	vR[i] = 81;
}
else
{
	vR[i] = 80;
}

}

}

}

}
else
{
	if(input[i]>(71+offset0))
{
	if(input[i]>(75+offset0))
{
	if(input[i]>(77+offset0))
{
	if(input[i]==(79+offset0))
{
	vR[i] = 79;
}
else
{
	vR[i] = 78;
}

}
else
{
	if(input[i]==(77+offset0))
{
	vR[i] = 77;
}
else
{
	vR[i] = 76;
}

}

}
else
{
	if(input[i]>(73+offset0))
{
	if(input[i]==(75+offset0))
{
	vR[i] = 75;
}
else
{
	vR[i] = 74;
}

}
else
{
	if(input[i]==(73+offset0))
{
	vR[i] = 73;
}
else
{
	vR[i] = 72;
}

}

}

}
else
{
	if(input[i]>(67+offset0))
{
	if(input[i]>(69+offset0))
{
	if(input[i]==(71+offset0))
{
	vR[i] = 71;
}
else
{
	vR[i] = 70;
}

}
else
{
	if(input[i]==(69+offset0))
{
	vR[i] = 69;
}
else
{
	vR[i] = 68;
}

}

}
else
{
	if(input[i]>(65+offset0))
{
	if(input[i]==(67+offset0))
{
	vR[i] = 67;
}
else
{
	vR[i] = 66;
}

}
else
{
	if(input[i]==(65+offset0))
{
	vR[i] = 65;
}
else
{
	vR[i] = 64;
}

}

}

}

}

}

}
else
{
	if(input[i]>(31+offset0))
{
	if(input[i]>(47+offset0))
{
	if(input[i]>(55+offset0))
{
	if(input[i]>(59+offset0))
{
	if(input[i]>(61+offset0))
{
	if(input[i]==(63+offset0))
{
	vR[i] = 63;
}
else
{
	vR[i] = 62;
}

}
else
{
	if(input[i]==(61+offset0))
{
	vR[i] = 61;
}
else
{
	vR[i] = 60;
}

}

}
else
{
	if(input[i]>(57+offset0))
{
	if(input[i]==(59+offset0))
{
	vR[i] = 59;
}
else
{
	vR[i] = 58;
}

}
else
{
	if(input[i]==(57+offset0))
{
	vR[i] = 57;
}
else
{
	vR[i] = 56;
}

}

}

}
else
{
	if(input[i]>(51+offset0))
{
	if(input[i]>(53+offset0))
{
	if(input[i]==(55+offset0))
{
	vR[i] = 55;
}
else
{
	vR[i] = 54;
}

}
else
{
	if(input[i]==(53+offset0))
{
	vR[i] = 53;
}
else
{
	vR[i] = 52;
}

}

}
else
{
	if(input[i]>(49+offset0))
{
	if(input[i]==(51+offset0))
{
	vR[i] = 51;
}
else
{
	vR[i] = 50;
}

}
else
{
	if(input[i]==(49+offset0))
{
	vR[i] = 49;
}
else
{
	vR[i] = 48;
}

}

}

}

}
else
{
	if(input[i]>(39+offset0))
{
	if(input[i]>(43+offset0))
{
	if(input[i]>(45+offset0))
{
	if(input[i]==(47+offset0))
{
	vR[i] = 47;
}
else
{
	vR[i] = 46;
}

}
else
{
	if(input[i]==(45+offset0))
{
	vR[i] = 45;
}
else
{
	vR[i] = 44;
}

}

}
else
{
	if(input[i]>(41+offset0))
{
	if(input[i]==(43+offset0))
{
	vR[i] = 43;
}
else
{
	vR[i] = 42;
}

}
else
{
	if(input[i]==(41+offset0))
{
	vR[i] = 41;
}
else
{
	vR[i] = 40;
}

}

}

}
else
{
	if(input[i]>(35+offset0))
{
	if(input[i]>(37+offset0))
{
	if(input[i]==(39+offset0))
{
	vR[i] = 39;
}
else
{
	vR[i] = 38;
}

}
else
{
	if(input[i]==(37+offset0))
{
	vR[i] = 37;
}
else
{
	vR[i] = 36;
}

}

}
else
{
	if(input[i]>(33+offset0))
{
	if(input[i]==(35+offset0))
{
	vR[i] = 35;
}
else
{
	vR[i] = 34;
}

}
else
{
	if(input[i]==(33+offset0))
{
	vR[i] = 33;
}
else
{
	vR[i] = 32;
}

}

}

}

}

}
else
{
	if(input[i]>(15+offset0))
{
	if(input[i]>(23+offset0))
{
	if(input[i]>(27+offset0))
{
	if(input[i]>(29+offset0))
{
	if(input[i]==(31+offset0))
{
	vR[i] = 31;
}
else
{
	vR[i] = 30;
}

}
else
{
	if(input[i]==(29+offset0))
{
	vR[i] = 29;
}
else
{
	vR[i] = 28;
}

}

}
else
{
	if(input[i]>(25+offset0))
{
	if(input[i]==(27+offset0))
{
	vR[i] = 27;
}
else
{
	vR[i] = 26;
}

}
else
{
	if(input[i]==(25+offset0))
{
	vR[i] = 25;
}
else
{
	vR[i] = 24;
}

}

}

}
else
{
	if(input[i]>(19+offset0))
{
	if(input[i]>(21+offset0))
{
	if(input[i]==(23+offset0))
{
	vR[i] = 23;
}
else
{
	vR[i] = 22;
}

}
else
{
	if(input[i]==(21+offset0))
{
	vR[i] = 21;
}
else
{
	vR[i] = 20;
}

}

}
else
{
	if(input[i]>(17+offset0))
{
	if(input[i]==(19+offset0))
{
	vR[i] = 19;
}
else
{
	vR[i] = 18;
}

}
else
{
	if(input[i]==(17+offset0))
{
	vR[i] = 17;
}
else
{
	vR[i] = 16;
}

}

}

}

}
else
{
	if(input[i]>(7+offset0))
{
	if(input[i]>(11+offset0))
{
	if(input[i]>(13+offset0))
{
	if(input[i]==(15+offset0))
{
	vR[i] = 15;
}
else
{
	vR[i] = 14;
}

}
else
{
	if(input[i]==(13+offset0))
{
	vR[i] = 13;
}
else
{
	vR[i] = 12;
}

}

}
else
{
	if(input[i]>(9+offset0))
{
	if(input[i]==(11+offset0))
{
	vR[i] = 11;
}
else
{
	vR[i] = 10;
}

}
else
{
	if(input[i]==(9+offset0))
{
	vR[i] = 9;
}
else
{
	vR[i] = 8;
}

}

}

}
else
{
	if(input[i]>(3+offset0))
{
	if(input[i]>(5+offset0))
{
	if(input[i]==(7+offset0))
{
	vR[i] = 7;
}
else
{
	vR[i] = 6;
}

}
else
{
	if(input[i]==(5+offset0))
{
	vR[i] = 5;
}
else
{
	vR[i] = 4;
}

}

}
else
{
	if(input[i]>(1+offset0))
{
	if(input[i]==(3+offset0))
{
	vR[i] = 3;
}
else
{
	vR[i] = 2;
}

}
else
{
	if(input[i]==(1+offset0))
{
	vR[i] = 1;
}
else
{
	vR[i] = 0;
}

}

}

}

}

}

}

}


	}
}

int main(int argc, char **argv)
{
	float elapsed_time;
	// set up device
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("[CUDA - Random Forest]: EXP %s\n", EXP_NAME);
	printf("[CUDA - Random Forest]: Using Device %d: %s\n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));

	// set up data size of vectors
	int nElem = N_ELEM;
	printf("[CUDA - Random Forest]: Vector Size %d\n", nElem);

	// malloc host memory
	size_t nBytes = nElem * sizeof(float);
	float *h_vR;
	h_vR = (float *)malloc(nBytes);
	float *h_input;
	h_input = (float *)malloc(nBytes);

	printf("[CUDA - Random Forest]: Start Reading Dataset\n");
	initData(h_input, nElem);
	printf("[CUDA - Random Forest]: Finished Reading Dataset\n");
	memset(h_vR, 0, nBytes);

	// malloc device global memory
	float *d_vR;
	CHECK(hipMalloc((float **)&d_vR, nBytes));
	float *d_input;
	CHECK(hipMalloc((float **)&d_input, nBytes));

	// transfer data from host to device
	CHECK(hipMemcpy(d_input, h_input, nBytes, hipMemcpyHostToDevice));

	// invoke kernel at host side
	int iLen = 512;
	dim3 block(iLen);
	dim3 grid((nElem + block.x - 1) / block.x);
	hipEvent_t start, stop;
	CHECK(hipEventCreate(&start));
	CHECK(hipEventCreate(&stop));

	// record start event
	CHECK(hipEventRecord(start, 0));
#if defined(_2TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, d_vR);
#elif defined(_3TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, d_vR);
#elif defined(_4TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, d_vR);
#else
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, d_vR);
#endif
	CHECK(hipEventRecord(stop, 0));
	CHECK(hipEventSynchronize(stop));
	hipDeviceSynchronize();

	// calculate elapsed time
	CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
	printf("[CUDA - Random Forest]: %s - execution time = %.6fms\n", EXP_NAME, elapsed_time);
	registerTime(elapsed_time);

	// copy kernel result back to host side
	CHECK(hipMemcpy(h_vR, d_vR, nBytes, hipMemcpyDeviceToHost));
	CHECK(hipGetLastError());

	//Verify the answers
	char flag = 1;
	for (int i = 0; i < nElem; i++)
	{
		if (i < 32)
		{
			printf("%.3f ", h_vR[i]);
		}
		if (h_input[i] != h_vR[i])
		{
			flag = 0;
			break;
		}
	}
	if (flag == 1)
	{
		printf("Return correct");
	}
	else
	{
		printf("Error!!");
	}
	printf("\n ");

	// free device memory
	hipFree(d_input);
	hipFree(d_vR);

	// free host memory
	free(h_input);
	free(h_vR);
	return (0);
}

void initData(float *ip, int size)
{
	for (int i = 0; i < size; i++)
	{

#if defined(_BEST)
		ip[i] = (float)(0);
#else
		if(WARP_SIZE < _MOD)
		{
			int aux = _MOD / WARP_SIZE;
			ip[i] = (float)((i * aux) % _MOD);
		}
		else
		{
			ip[i] = (float)(i % _MOD);
		}
#endif
	}
	return;
}

void registerTime(float value)
{
	printf("[CUDA - Random Forest]: Time spent: %f.\n", value);
}
