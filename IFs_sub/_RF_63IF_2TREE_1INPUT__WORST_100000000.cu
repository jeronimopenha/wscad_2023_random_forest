#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>

#define _WORST
#define N_ELEM 100000000

#define WARP_SIZE 32

#define CHECK(call)                                                \
	{                                                              \
		const hipError_t error = call;                            \
		if (error != hipSuccess)                                  \
		{                                                          \
			fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
			fprintf(stderr, "code: %d, reason: %s\n", error,       \
					hipGetErrorString(error));                    \
		}                                                          \
	}

void initData(float *ip, int size);
void registerTime(float value);

#define _MOD 64
#define _2TREE
#define EXP_NAME "_RF_63IF_2TREE_1INPUT__WORST_100000000"

// RF_with_if

__global__ void RF_kernel(float *input, const int n, const int offset0, const int offset1, float *vR)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
if(input[i]>(31+offset0))
{
	if(input[i]>(47+offset0))
{
	if(input[i]>(55+offset0))
{
	if(input[i]>(59+offset0))
{
	if(input[i]>(61+offset0))
{
	if(input[i]==(63+offset0))
{
	vR[i] = 63;
}
else
{
	vR[i] = 62;
}

}
else
{
	if(input[i]==(61+offset0))
{
	vR[i] = 61;
}
else
{
	vR[i] = 60;
}

}

}
else
{
	if(input[i]>(57+offset0))
{
	if(input[i]==(59+offset0))
{
	vR[i] = 59;
}
else
{
	vR[i] = 58;
}

}
else
{
	if(input[i]==(57+offset0))
{
	vR[i] = 57;
}
else
{
	vR[i] = 56;
}

}

}

}
else
{
	if(input[i]>(51+offset0))
{
	if(input[i]>(53+offset0))
{
	if(input[i]==(55+offset0))
{
	vR[i] = 55;
}
else
{
	vR[i] = 54;
}

}
else
{
	if(input[i]==(53+offset0))
{
	vR[i] = 53;
}
else
{
	vR[i] = 52;
}

}

}
else
{
	if(input[i]>(49+offset0))
{
	if(input[i]==(51+offset0))
{
	vR[i] = 51;
}
else
{
	vR[i] = 50;
}

}
else
{
	if(input[i]==(49+offset0))
{
	vR[i] = 49;
}
else
{
	vR[i] = 48;
}

}

}

}

}
else
{
	if(input[i]>(39+offset0))
{
	if(input[i]>(43+offset0))
{
	if(input[i]>(45+offset0))
{
	if(input[i]==(47+offset0))
{
	vR[i] = 47;
}
else
{
	vR[i] = 46;
}

}
else
{
	if(input[i]==(45+offset0))
{
	vR[i] = 45;
}
else
{
	vR[i] = 44;
}

}

}
else
{
	if(input[i]>(41+offset0))
{
	if(input[i]==(43+offset0))
{
	vR[i] = 43;
}
else
{
	vR[i] = 42;
}

}
else
{
	if(input[i]==(41+offset0))
{
	vR[i] = 41;
}
else
{
	vR[i] = 40;
}

}

}

}
else
{
	if(input[i]>(35+offset0))
{
	if(input[i]>(37+offset0))
{
	if(input[i]==(39+offset0))
{
	vR[i] = 39;
}
else
{
	vR[i] = 38;
}

}
else
{
	if(input[i]==(37+offset0))
{
	vR[i] = 37;
}
else
{
	vR[i] = 36;
}

}

}
else
{
	if(input[i]>(33+offset0))
{
	if(input[i]==(35+offset0))
{
	vR[i] = 35;
}
else
{
	vR[i] = 34;
}

}
else
{
	if(input[i]==(33+offset0))
{
	vR[i] = 33;
}
else
{
	vR[i] = 32;
}

}

}

}

}

}
else
{
	if(input[i]>(15+offset0))
{
	if(input[i]>(23+offset0))
{
	if(input[i]>(27+offset0))
{
	if(input[i]>(29+offset0))
{
	if(input[i]==(31+offset0))
{
	vR[i] = 31;
}
else
{
	vR[i] = 30;
}

}
else
{
	if(input[i]==(29+offset0))
{
	vR[i] = 29;
}
else
{
	vR[i] = 28;
}

}

}
else
{
	if(input[i]>(25+offset0))
{
	if(input[i]==(27+offset0))
{
	vR[i] = 27;
}
else
{
	vR[i] = 26;
}

}
else
{
	if(input[i]==(25+offset0))
{
	vR[i] = 25;
}
else
{
	vR[i] = 24;
}

}

}

}
else
{
	if(input[i]>(19+offset0))
{
	if(input[i]>(21+offset0))
{
	if(input[i]==(23+offset0))
{
	vR[i] = 23;
}
else
{
	vR[i] = 22;
}

}
else
{
	if(input[i]==(21+offset0))
{
	vR[i] = 21;
}
else
{
	vR[i] = 20;
}

}

}
else
{
	if(input[i]>(17+offset0))
{
	if(input[i]==(19+offset0))
{
	vR[i] = 19;
}
else
{
	vR[i] = 18;
}

}
else
{
	if(input[i]==(17+offset0))
{
	vR[i] = 17;
}
else
{
	vR[i] = 16;
}

}

}

}

}
else
{
	if(input[i]>(7+offset0))
{
	if(input[i]>(11+offset0))
{
	if(input[i]>(13+offset0))
{
	if(input[i]==(15+offset0))
{
	vR[i] = 15;
}
else
{
	vR[i] = 14;
}

}
else
{
	if(input[i]==(13+offset0))
{
	vR[i] = 13;
}
else
{
	vR[i] = 12;
}

}

}
else
{
	if(input[i]>(9+offset0))
{
	if(input[i]==(11+offset0))
{
	vR[i] = 11;
}
else
{
	vR[i] = 10;
}

}
else
{
	if(input[i]==(9+offset0))
{
	vR[i] = 9;
}
else
{
	vR[i] = 8;
}

}

}

}
else
{
	if(input[i]>(3+offset0))
{
	if(input[i]>(5+offset0))
{
	if(input[i]==(7+offset0))
{
	vR[i] = 7;
}
else
{
	vR[i] = 6;
}

}
else
{
	if(input[i]==(5+offset0))
{
	vR[i] = 5;
}
else
{
	vR[i] = 4;
}

}

}
else
{
	if(input[i]>(1+offset0))
{
	if(input[i]==(3+offset0))
{
	vR[i] = 3;
}
else
{
	vR[i] = 2;
}

}
else
{
	if(input[i]==(1+offset0))
{
	vR[i] = 1;
}
else
{
	vR[i] = 0;
}

}

}

}

}

}


if(input[i]>(31+offset1))
{
	if(input[i]>(47+offset1))
{
	if(input[i]>(55+offset1))
{
	if(input[i]>(59+offset1))
{
	if(input[i]>(61+offset1))
{
	if(input[i]==(63+offset1))
{
	vR[i] = 63;
}
else
{
	vR[i] = 62;
}

}
else
{
	if(input[i]==(61+offset1))
{
	vR[i] = 61;
}
else
{
	vR[i] = 60;
}

}

}
else
{
	if(input[i]>(57+offset1))
{
	if(input[i]==(59+offset1))
{
	vR[i] = 59;
}
else
{
	vR[i] = 58;
}

}
else
{
	if(input[i]==(57+offset1))
{
	vR[i] = 57;
}
else
{
	vR[i] = 56;
}

}

}

}
else
{
	if(input[i]>(51+offset1))
{
	if(input[i]>(53+offset1))
{
	if(input[i]==(55+offset1))
{
	vR[i] = 55;
}
else
{
	vR[i] = 54;
}

}
else
{
	if(input[i]==(53+offset1))
{
	vR[i] = 53;
}
else
{
	vR[i] = 52;
}

}

}
else
{
	if(input[i]>(49+offset1))
{
	if(input[i]==(51+offset1))
{
	vR[i] = 51;
}
else
{
	vR[i] = 50;
}

}
else
{
	if(input[i]==(49+offset1))
{
	vR[i] = 49;
}
else
{
	vR[i] = 48;
}

}

}

}

}
else
{
	if(input[i]>(39+offset1))
{
	if(input[i]>(43+offset1))
{
	if(input[i]>(45+offset1))
{
	if(input[i]==(47+offset1))
{
	vR[i] = 47;
}
else
{
	vR[i] = 46;
}

}
else
{
	if(input[i]==(45+offset1))
{
	vR[i] = 45;
}
else
{
	vR[i] = 44;
}

}

}
else
{
	if(input[i]>(41+offset1))
{
	if(input[i]==(43+offset1))
{
	vR[i] = 43;
}
else
{
	vR[i] = 42;
}

}
else
{
	if(input[i]==(41+offset1))
{
	vR[i] = 41;
}
else
{
	vR[i] = 40;
}

}

}

}
else
{
	if(input[i]>(35+offset1))
{
	if(input[i]>(37+offset1))
{
	if(input[i]==(39+offset1))
{
	vR[i] = 39;
}
else
{
	vR[i] = 38;
}

}
else
{
	if(input[i]==(37+offset1))
{
	vR[i] = 37;
}
else
{
	vR[i] = 36;
}

}

}
else
{
	if(input[i]>(33+offset1))
{
	if(input[i]==(35+offset1))
{
	vR[i] = 35;
}
else
{
	vR[i] = 34;
}

}
else
{
	if(input[i]==(33+offset1))
{
	vR[i] = 33;
}
else
{
	vR[i] = 32;
}

}

}

}

}

}
else
{
	if(input[i]>(15+offset1))
{
	if(input[i]>(23+offset1))
{
	if(input[i]>(27+offset1))
{
	if(input[i]>(29+offset1))
{
	if(input[i]==(31+offset1))
{
	vR[i] = 31;
}
else
{
	vR[i] = 30;
}

}
else
{
	if(input[i]==(29+offset1))
{
	vR[i] = 29;
}
else
{
	vR[i] = 28;
}

}

}
else
{
	if(input[i]>(25+offset1))
{
	if(input[i]==(27+offset1))
{
	vR[i] = 27;
}
else
{
	vR[i] = 26;
}

}
else
{
	if(input[i]==(25+offset1))
{
	vR[i] = 25;
}
else
{
	vR[i] = 24;
}

}

}

}
else
{
	if(input[i]>(19+offset1))
{
	if(input[i]>(21+offset1))
{
	if(input[i]==(23+offset1))
{
	vR[i] = 23;
}
else
{
	vR[i] = 22;
}

}
else
{
	if(input[i]==(21+offset1))
{
	vR[i] = 21;
}
else
{
	vR[i] = 20;
}

}

}
else
{
	if(input[i]>(17+offset1))
{
	if(input[i]==(19+offset1))
{
	vR[i] = 19;
}
else
{
	vR[i] = 18;
}

}
else
{
	if(input[i]==(17+offset1))
{
	vR[i] = 17;
}
else
{
	vR[i] = 16;
}

}

}

}

}
else
{
	if(input[i]>(7+offset1))
{
	if(input[i]>(11+offset1))
{
	if(input[i]>(13+offset1))
{
	if(input[i]==(15+offset1))
{
	vR[i] = 15;
}
else
{
	vR[i] = 14;
}

}
else
{
	if(input[i]==(13+offset1))
{
	vR[i] = 13;
}
else
{
	vR[i] = 12;
}

}

}
else
{
	if(input[i]>(9+offset1))
{
	if(input[i]==(11+offset1))
{
	vR[i] = 11;
}
else
{
	vR[i] = 10;
}

}
else
{
	if(input[i]==(9+offset1))
{
	vR[i] = 9;
}
else
{
	vR[i] = 8;
}

}

}

}
else
{
	if(input[i]>(3+offset1))
{
	if(input[i]>(5+offset1))
{
	if(input[i]==(7+offset1))
{
	vR[i] = 7;
}
else
{
	vR[i] = 6;
}

}
else
{
	if(input[i]==(5+offset1))
{
	vR[i] = 5;
}
else
{
	vR[i] = 4;
}

}

}
else
{
	if(input[i]>(1+offset1))
{
	if(input[i]==(3+offset1))
{
	vR[i] = 3;
}
else
{
	vR[i] = 2;
}

}
else
{
	if(input[i]==(1+offset1))
{
	vR[i] = 1;
}
else
{
	vR[i] = 0;
}

}

}

}

}

}


	}
}

int main(int argc, char **argv)
{
	float elapsed_time;
	// set up device
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("[CUDA - Random Forest]: EXP %s\n", EXP_NAME);
	printf("[CUDA - Random Forest]: Using Device %d: %s\n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));

	// set up data size of vectors
	int nElem = N_ELEM;
	printf("[CUDA - Random Forest]: Vector Size %d\n", nElem);

	// malloc host memory
	size_t nBytes = nElem * sizeof(float);
	float *h_vR;
	h_vR = (float *)malloc(nBytes);
	float *h_input;
	h_input = (float *)malloc(nBytes);

	printf("[CUDA - Random Forest]: Start Reading Dataset\n");
	initData(h_input, nElem);
	printf("[CUDA - Random Forest]: Finished Reading Dataset\n");
	memset(h_vR, 0, nBytes);

	// malloc device global memory
	float *d_vR;
	CHECK(hipMalloc((float **)&d_vR, nBytes));
	float *d_input;
	CHECK(hipMalloc((float **)&d_input, nBytes));

	// transfer data from host to device
	CHECK(hipMemcpy(d_input, h_input, nBytes, hipMemcpyHostToDevice));

	// invoke kernel at host side
	int iLen = 512;
	dim3 block(iLen);
	dim3 grid((nElem + block.x - 1) / block.x);
	hipEvent_t start, stop;
	CHECK(hipEventCreate(&start));
	CHECK(hipEventCreate(&stop));

	// record start event
	CHECK(hipEventRecord(start, 0));
#if defined(_2TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, d_vR);
#elif defined(_3TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, d_vR);
#elif defined(_4TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, d_vR);
#else
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, d_vR);
#endif
	CHECK(hipEventRecord(stop, 0));
	CHECK(hipEventSynchronize(stop));
	hipDeviceSynchronize();

	// calculate elapsed time
	CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
	printf("[CUDA - Random Forest]: %s - execution time = %.6fms\n", EXP_NAME, elapsed_time);
	registerTime(elapsed_time);

	// copy kernel result back to host side
	CHECK(hipMemcpy(h_vR, d_vR, nBytes, hipMemcpyDeviceToHost));
	CHECK(hipGetLastError());

	//Verify the answers
	char flag = 1;
	for (int i = 0; i < nElem; i++)
	{
		if (i < 32)
		{
			printf("%.3f ", h_vR[i]);
		}
		if (h_input[i] != h_vR[i])
		{
			flag = 0;
			break;
		}
	}
	if (flag == 1)
	{
		printf("Return correct");
	}
	else
	{
		printf("Error!!");
	}
	printf("\n ");

	// free device memory
	hipFree(d_input);
	hipFree(d_vR);

	// free host memory
	free(h_input);
	free(h_vR);
	return (0);
}

void initData(float *ip, int size)
{
	for (int i = 0; i < size; i++)
	{

#if defined(_BEST)
		ip[i] = (float)(0);
#else
		if(WARP_SIZE < _MOD)
		{
			int aux = _MOD / WARP_SIZE;
			ip[i] = (float)((i * aux) % _MOD);
		}
		else
		{
			ip[i] = (float)(i % _MOD);
		}
#endif
	}
	return;
}

void registerTime(float value)
{
	printf("[CUDA - Random Forest]: Time spent: %f.\n", value);
}
