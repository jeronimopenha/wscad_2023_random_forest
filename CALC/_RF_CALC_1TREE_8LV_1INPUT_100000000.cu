#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>

#define N_ELEM 100000000

#define WARP_SIZE 32

#define CHECK(call)                                                \
	{                                                              \
		const hipError_t error = call;                            \
		if (error != hipSuccess)                                  \
		{                                                          \
			fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
			fprintf(stderr, "code: %d, reason: %s\n", error,       \
					hipGetErrorString(error));                    \
		}                                                          \
	}

void initData(float *ip, int size);
void registerTime(float value);

#define _MOD 256
#define _1TREE
#define EXP_NAME "_RF_CALC_1TREE_8LV_1INPUT_100000000"

// RF_with_if

__global__ void RF_kernel(float *input, const int n, const int offset0, float *vR)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
int rt0_0_0 = (input[i] > (1+offset0));
int t0_0_0 = rt0_0_0 * (2 + (input[i] > (2+offset0)));
t0_0_0 += (1 - rt0_0_0) * (input[i] > (0+offset0));

int rt0_0_1 = (input[i] > (5+offset0));
int t0_0_1 = rt0_0_1 * (2 + (input[i] > (6+offset0)));
t0_0_1 += (1 - rt0_0_1) * (input[i] > (4+offset0));

int rt0_0_2 = (input[i] > (9+offset0));
int t0_0_2 = rt0_0_2 * (2 + (input[i] > (10+offset0)));
t0_0_2 += (1 - rt0_0_2) * (input[i] > (8+offset0));

int rt0_0_3 = (input[i] > (13+offset0));
int t0_0_3 = rt0_0_3 * (2 + (input[i] > (14+offset0)));
t0_0_3 += (1 - rt0_0_3) * (input[i] > (12+offset0));

int rt0_0_4 = (input[i] > (17+offset0));
int t0_0_4 = rt0_0_4 * (2 + (input[i] > (18+offset0)));
t0_0_4 += (1 - rt0_0_4) * (input[i] > (16+offset0));

int rt0_0_5 = (input[i] > (21+offset0));
int t0_0_5 = rt0_0_5 * (2 + (input[i] > (22+offset0)));
t0_0_5 += (1 - rt0_0_5) * (input[i] > (20+offset0));

int rt0_0_6 = (input[i] > (25+offset0));
int t0_0_6 = rt0_0_6 * (2 + (input[i] > (26+offset0)));
t0_0_6 += (1 - rt0_0_6) * (input[i] > (24+offset0));

int rt0_0_7 = (input[i] > (29+offset0));
int t0_0_7 = rt0_0_7 * (2 + (input[i] > (30+offset0)));
t0_0_7 += (1 - rt0_0_7) * (input[i] > (28+offset0));

int rt0_0_8 = (input[i] > (33+offset0));
int t0_0_8 = rt0_0_8 * (2 + (input[i] > (34+offset0)));
t0_0_8 += (1 - rt0_0_8) * (input[i] > (32+offset0));

int rt0_0_9 = (input[i] > (37+offset0));
int t0_0_9 = rt0_0_9 * (2 + (input[i] > (38+offset0)));
t0_0_9 += (1 - rt0_0_9) * (input[i] > (36+offset0));

int rt0_0_10 = (input[i] > (41+offset0));
int t0_0_10 = rt0_0_10 * (2 + (input[i] > (42+offset0)));
t0_0_10 += (1 - rt0_0_10) * (input[i] > (40+offset0));

int rt0_0_11 = (input[i] > (45+offset0));
int t0_0_11 = rt0_0_11 * (2 + (input[i] > (46+offset0)));
t0_0_11 += (1 - rt0_0_11) * (input[i] > (44+offset0));

int rt0_0_12 = (input[i] > (49+offset0));
int t0_0_12 = rt0_0_12 * (2 + (input[i] > (50+offset0)));
t0_0_12 += (1 - rt0_0_12) * (input[i] > (48+offset0));

int rt0_0_13 = (input[i] > (53+offset0));
int t0_0_13 = rt0_0_13 * (2 + (input[i] > (54+offset0)));
t0_0_13 += (1 - rt0_0_13) * (input[i] > (52+offset0));

int rt0_0_14 = (input[i] > (57+offset0));
int t0_0_14 = rt0_0_14 * (2 + (input[i] > (58+offset0)));
t0_0_14 += (1 - rt0_0_14) * (input[i] > (56+offset0));

int rt0_0_15 = (input[i] > (61+offset0));
int t0_0_15 = rt0_0_15 * (2 + (input[i] > (62+offset0)));
t0_0_15 += (1 - rt0_0_15) * (input[i] > (60+offset0));

int rt0_0_16 = (input[i] > (65+offset0));
int t0_0_16 = rt0_0_16 * (2 + (input[i] > (66+offset0)));
t0_0_16 += (1 - rt0_0_16) * (input[i] > (64+offset0));

int rt0_0_17 = (input[i] > (69+offset0));
int t0_0_17 = rt0_0_17 * (2 + (input[i] > (70+offset0)));
t0_0_17 += (1 - rt0_0_17) * (input[i] > (68+offset0));

int rt0_0_18 = (input[i] > (73+offset0));
int t0_0_18 = rt0_0_18 * (2 + (input[i] > (74+offset0)));
t0_0_18 += (1 - rt0_0_18) * (input[i] > (72+offset0));

int rt0_0_19 = (input[i] > (77+offset0));
int t0_0_19 = rt0_0_19 * (2 + (input[i] > (78+offset0)));
t0_0_19 += (1 - rt0_0_19) * (input[i] > (76+offset0));

int rt0_0_20 = (input[i] > (81+offset0));
int t0_0_20 = rt0_0_20 * (2 + (input[i] > (82+offset0)));
t0_0_20 += (1 - rt0_0_20) * (input[i] > (80+offset0));

int rt0_0_21 = (input[i] > (85+offset0));
int t0_0_21 = rt0_0_21 * (2 + (input[i] > (86+offset0)));
t0_0_21 += (1 - rt0_0_21) * (input[i] > (84+offset0));

int rt0_0_22 = (input[i] > (89+offset0));
int t0_0_22 = rt0_0_22 * (2 + (input[i] > (90+offset0)));
t0_0_22 += (1 - rt0_0_22) * (input[i] > (88+offset0));

int rt0_0_23 = (input[i] > (93+offset0));
int t0_0_23 = rt0_0_23 * (2 + (input[i] > (94+offset0)));
t0_0_23 += (1 - rt0_0_23) * (input[i] > (92+offset0));

int rt0_0_24 = (input[i] > (97+offset0));
int t0_0_24 = rt0_0_24 * (2 + (input[i] > (98+offset0)));
t0_0_24 += (1 - rt0_0_24) * (input[i] > (96+offset0));

int rt0_0_25 = (input[i] > (101+offset0));
int t0_0_25 = rt0_0_25 * (2 + (input[i] > (102+offset0)));
t0_0_25 += (1 - rt0_0_25) * (input[i] > (100+offset0));

int rt0_0_26 = (input[i] > (105+offset0));
int t0_0_26 = rt0_0_26 * (2 + (input[i] > (106+offset0)));
t0_0_26 += (1 - rt0_0_26) * (input[i] > (104+offset0));

int rt0_0_27 = (input[i] > (109+offset0));
int t0_0_27 = rt0_0_27 * (2 + (input[i] > (110+offset0)));
t0_0_27 += (1 - rt0_0_27) * (input[i] > (108+offset0));

int rt0_0_28 = (input[i] > (113+offset0));
int t0_0_28 = rt0_0_28 * (2 + (input[i] > (114+offset0)));
t0_0_28 += (1 - rt0_0_28) * (input[i] > (112+offset0));

int rt0_0_29 = (input[i] > (117+offset0));
int t0_0_29 = rt0_0_29 * (2 + (input[i] > (118+offset0)));
t0_0_29 += (1 - rt0_0_29) * (input[i] > (116+offset0));

int rt0_0_30 = (input[i] > (121+offset0));
int t0_0_30 = rt0_0_30 * (2 + (input[i] > (122+offset0)));
t0_0_30 += (1 - rt0_0_30) * (input[i] > (120+offset0));

int rt0_0_31 = (input[i] > (125+offset0));
int t0_0_31 = rt0_0_31 * (2 + (input[i] > (126+offset0)));
t0_0_31 += (1 - rt0_0_31) * (input[i] > (124+offset0));

int rt0_0_32 = (input[i] > (129+offset0));
int t0_0_32 = rt0_0_32 * (2 + (input[i] > (130+offset0)));
t0_0_32 += (1 - rt0_0_32) * (input[i] > (128+offset0));

int rt0_0_33 = (input[i] > (133+offset0));
int t0_0_33 = rt0_0_33 * (2 + (input[i] > (134+offset0)));
t0_0_33 += (1 - rt0_0_33) * (input[i] > (132+offset0));

int rt0_0_34 = (input[i] > (137+offset0));
int t0_0_34 = rt0_0_34 * (2 + (input[i] > (138+offset0)));
t0_0_34 += (1 - rt0_0_34) * (input[i] > (136+offset0));

int rt0_0_35 = (input[i] > (141+offset0));
int t0_0_35 = rt0_0_35 * (2 + (input[i] > (142+offset0)));
t0_0_35 += (1 - rt0_0_35) * (input[i] > (140+offset0));

int rt0_0_36 = (input[i] > (145+offset0));
int t0_0_36 = rt0_0_36 * (2 + (input[i] > (146+offset0)));
t0_0_36 += (1 - rt0_0_36) * (input[i] > (144+offset0));

int rt0_0_37 = (input[i] > (149+offset0));
int t0_0_37 = rt0_0_37 * (2 + (input[i] > (150+offset0)));
t0_0_37 += (1 - rt0_0_37) * (input[i] > (148+offset0));

int rt0_0_38 = (input[i] > (153+offset0));
int t0_0_38 = rt0_0_38 * (2 + (input[i] > (154+offset0)));
t0_0_38 += (1 - rt0_0_38) * (input[i] > (152+offset0));

int rt0_0_39 = (input[i] > (157+offset0));
int t0_0_39 = rt0_0_39 * (2 + (input[i] > (158+offset0)));
t0_0_39 += (1 - rt0_0_39) * (input[i] > (156+offset0));

int rt0_0_40 = (input[i] > (161+offset0));
int t0_0_40 = rt0_0_40 * (2 + (input[i] > (162+offset0)));
t0_0_40 += (1 - rt0_0_40) * (input[i] > (160+offset0));

int rt0_0_41 = (input[i] > (165+offset0));
int t0_0_41 = rt0_0_41 * (2 + (input[i] > (166+offset0)));
t0_0_41 += (1 - rt0_0_41) * (input[i] > (164+offset0));

int rt0_0_42 = (input[i] > (169+offset0));
int t0_0_42 = rt0_0_42 * (2 + (input[i] > (170+offset0)));
t0_0_42 += (1 - rt0_0_42) * (input[i] > (168+offset0));

int rt0_0_43 = (input[i] > (173+offset0));
int t0_0_43 = rt0_0_43 * (2 + (input[i] > (174+offset0)));
t0_0_43 += (1 - rt0_0_43) * (input[i] > (172+offset0));

int rt0_0_44 = (input[i] > (177+offset0));
int t0_0_44 = rt0_0_44 * (2 + (input[i] > (178+offset0)));
t0_0_44 += (1 - rt0_0_44) * (input[i] > (176+offset0));

int rt0_0_45 = (input[i] > (181+offset0));
int t0_0_45 = rt0_0_45 * (2 + (input[i] > (182+offset0)));
t0_0_45 += (1 - rt0_0_45) * (input[i] > (180+offset0));

int rt0_0_46 = (input[i] > (185+offset0));
int t0_0_46 = rt0_0_46 * (2 + (input[i] > (186+offset0)));
t0_0_46 += (1 - rt0_0_46) * (input[i] > (184+offset0));

int rt0_0_47 = (input[i] > (189+offset0));
int t0_0_47 = rt0_0_47 * (2 + (input[i] > (190+offset0)));
t0_0_47 += (1 - rt0_0_47) * (input[i] > (188+offset0));

int rt0_0_48 = (input[i] > (193+offset0));
int t0_0_48 = rt0_0_48 * (2 + (input[i] > (194+offset0)));
t0_0_48 += (1 - rt0_0_48) * (input[i] > (192+offset0));

int rt0_0_49 = (input[i] > (197+offset0));
int t0_0_49 = rt0_0_49 * (2 + (input[i] > (198+offset0)));
t0_0_49 += (1 - rt0_0_49) * (input[i] > (196+offset0));

int rt0_0_50 = (input[i] > (201+offset0));
int t0_0_50 = rt0_0_50 * (2 + (input[i] > (202+offset0)));
t0_0_50 += (1 - rt0_0_50) * (input[i] > (200+offset0));

int rt0_0_51 = (input[i] > (205+offset0));
int t0_0_51 = rt0_0_51 * (2 + (input[i] > (206+offset0)));
t0_0_51 += (1 - rt0_0_51) * (input[i] > (204+offset0));

int rt0_0_52 = (input[i] > (209+offset0));
int t0_0_52 = rt0_0_52 * (2 + (input[i] > (210+offset0)));
t0_0_52 += (1 - rt0_0_52) * (input[i] > (208+offset0));

int rt0_0_53 = (input[i] > (213+offset0));
int t0_0_53 = rt0_0_53 * (2 + (input[i] > (214+offset0)));
t0_0_53 += (1 - rt0_0_53) * (input[i] > (212+offset0));

int rt0_0_54 = (input[i] > (217+offset0));
int t0_0_54 = rt0_0_54 * (2 + (input[i] > (218+offset0)));
t0_0_54 += (1 - rt0_0_54) * (input[i] > (216+offset0));

int rt0_0_55 = (input[i] > (221+offset0));
int t0_0_55 = rt0_0_55 * (2 + (input[i] > (222+offset0)));
t0_0_55 += (1 - rt0_0_55) * (input[i] > (220+offset0));

int rt0_0_56 = (input[i] > (225+offset0));
int t0_0_56 = rt0_0_56 * (2 + (input[i] > (226+offset0)));
t0_0_56 += (1 - rt0_0_56) * (input[i] > (224+offset0));

int rt0_0_57 = (input[i] > (229+offset0));
int t0_0_57 = rt0_0_57 * (2 + (input[i] > (230+offset0)));
t0_0_57 += (1 - rt0_0_57) * (input[i] > (228+offset0));

int rt0_0_58 = (input[i] > (233+offset0));
int t0_0_58 = rt0_0_58 * (2 + (input[i] > (234+offset0)));
t0_0_58 += (1 - rt0_0_58) * (input[i] > (232+offset0));

int rt0_0_59 = (input[i] > (237+offset0));
int t0_0_59 = rt0_0_59 * (2 + (input[i] > (238+offset0)));
t0_0_59 += (1 - rt0_0_59) * (input[i] > (236+offset0));

int rt0_0_60 = (input[i] > (241+offset0));
int t0_0_60 = rt0_0_60 * (2 + (input[i] > (242+offset0)));
t0_0_60 += (1 - rt0_0_60) * (input[i] > (240+offset0));

int rt0_0_61 = (input[i] > (245+offset0));
int t0_0_61 = rt0_0_61 * (2 + (input[i] > (246+offset0)));
t0_0_61 += (1 - rt0_0_61) * (input[i] > (244+offset0));

int rt0_0_62 = (input[i] > (249+offset0));
int t0_0_62 = rt0_0_62 * (2 + (input[i] > (250+offset0)));
t0_0_62 += (1 - rt0_0_62) * (input[i] > (248+offset0));

int rt0_0_63 = (input[i] > (253+offset0));
int t0_0_63 = rt0_0_63 * (2 + (input[i] > (254+offset0)));
t0_0_63 += (1 - rt0_0_63) * (input[i] > (252+offset0));

int rt0_1_0 = (input[i] > (3+offset0));
int t0_1_0 = rt0_1_0 * (4 + t0_0_1);
t0_1_0 += (1 - rt0_1_0) * t0_0_0;

int rt0_1_1 = (input[i] > (11+offset0));
int t0_1_1 = rt0_1_1 * (4 + t0_0_3);
t0_1_1 += (1 - rt0_1_1) * t0_0_2;

int rt0_1_2 = (input[i] > (19+offset0));
int t0_1_2 = rt0_1_2 * (4 + t0_0_5);
t0_1_2 += (1 - rt0_1_2) * t0_0_4;

int rt0_1_3 = (input[i] > (27+offset0));
int t0_1_3 = rt0_1_3 * (4 + t0_0_7);
t0_1_3 += (1 - rt0_1_3) * t0_0_6;

int rt0_1_4 = (input[i] > (35+offset0));
int t0_1_4 = rt0_1_4 * (4 + t0_0_9);
t0_1_4 += (1 - rt0_1_4) * t0_0_8;

int rt0_1_5 = (input[i] > (43+offset0));
int t0_1_5 = rt0_1_5 * (4 + t0_0_11);
t0_1_5 += (1 - rt0_1_5) * t0_0_10;

int rt0_1_6 = (input[i] > (51+offset0));
int t0_1_6 = rt0_1_6 * (4 + t0_0_13);
t0_1_6 += (1 - rt0_1_6) * t0_0_12;

int rt0_1_7 = (input[i] > (59+offset0));
int t0_1_7 = rt0_1_7 * (4 + t0_0_15);
t0_1_7 += (1 - rt0_1_7) * t0_0_14;

int rt0_1_8 = (input[i] > (67+offset0));
int t0_1_8 = rt0_1_8 * (4 + t0_0_17);
t0_1_8 += (1 - rt0_1_8) * t0_0_16;

int rt0_1_9 = (input[i] > (75+offset0));
int t0_1_9 = rt0_1_9 * (4 + t0_0_19);
t0_1_9 += (1 - rt0_1_9) * t0_0_18;

int rt0_1_10 = (input[i] > (83+offset0));
int t0_1_10 = rt0_1_10 * (4 + t0_0_21);
t0_1_10 += (1 - rt0_1_10) * t0_0_20;

int rt0_1_11 = (input[i] > (91+offset0));
int t0_1_11 = rt0_1_11 * (4 + t0_0_23);
t0_1_11 += (1 - rt0_1_11) * t0_0_22;

int rt0_1_12 = (input[i] > (99+offset0));
int t0_1_12 = rt0_1_12 * (4 + t0_0_25);
t0_1_12 += (1 - rt0_1_12) * t0_0_24;

int rt0_1_13 = (input[i] > (107+offset0));
int t0_1_13 = rt0_1_13 * (4 + t0_0_27);
t0_1_13 += (1 - rt0_1_13) * t0_0_26;

int rt0_1_14 = (input[i] > (115+offset0));
int t0_1_14 = rt0_1_14 * (4 + t0_0_29);
t0_1_14 += (1 - rt0_1_14) * t0_0_28;

int rt0_1_15 = (input[i] > (123+offset0));
int t0_1_15 = rt0_1_15 * (4 + t0_0_31);
t0_1_15 += (1 - rt0_1_15) * t0_0_30;

int rt0_1_16 = (input[i] > (131+offset0));
int t0_1_16 = rt0_1_16 * (4 + t0_0_33);
t0_1_16 += (1 - rt0_1_16) * t0_0_32;

int rt0_1_17 = (input[i] > (139+offset0));
int t0_1_17 = rt0_1_17 * (4 + t0_0_35);
t0_1_17 += (1 - rt0_1_17) * t0_0_34;

int rt0_1_18 = (input[i] > (147+offset0));
int t0_1_18 = rt0_1_18 * (4 + t0_0_37);
t0_1_18 += (1 - rt0_1_18) * t0_0_36;

int rt0_1_19 = (input[i] > (155+offset0));
int t0_1_19 = rt0_1_19 * (4 + t0_0_39);
t0_1_19 += (1 - rt0_1_19) * t0_0_38;

int rt0_1_20 = (input[i] > (163+offset0));
int t0_1_20 = rt0_1_20 * (4 + t0_0_41);
t0_1_20 += (1 - rt0_1_20) * t0_0_40;

int rt0_1_21 = (input[i] > (171+offset0));
int t0_1_21 = rt0_1_21 * (4 + t0_0_43);
t0_1_21 += (1 - rt0_1_21) * t0_0_42;

int rt0_1_22 = (input[i] > (179+offset0));
int t0_1_22 = rt0_1_22 * (4 + t0_0_45);
t0_1_22 += (1 - rt0_1_22) * t0_0_44;

int rt0_1_23 = (input[i] > (187+offset0));
int t0_1_23 = rt0_1_23 * (4 + t0_0_47);
t0_1_23 += (1 - rt0_1_23) * t0_0_46;

int rt0_1_24 = (input[i] > (195+offset0));
int t0_1_24 = rt0_1_24 * (4 + t0_0_49);
t0_1_24 += (1 - rt0_1_24) * t0_0_48;

int rt0_1_25 = (input[i] > (203+offset0));
int t0_1_25 = rt0_1_25 * (4 + t0_0_51);
t0_1_25 += (1 - rt0_1_25) * t0_0_50;

int rt0_1_26 = (input[i] > (211+offset0));
int t0_1_26 = rt0_1_26 * (4 + t0_0_53);
t0_1_26 += (1 - rt0_1_26) * t0_0_52;

int rt0_1_27 = (input[i] > (219+offset0));
int t0_1_27 = rt0_1_27 * (4 + t0_0_55);
t0_1_27 += (1 - rt0_1_27) * t0_0_54;

int rt0_1_28 = (input[i] > (227+offset0));
int t0_1_28 = rt0_1_28 * (4 + t0_0_57);
t0_1_28 += (1 - rt0_1_28) * t0_0_56;

int rt0_1_29 = (input[i] > (235+offset0));
int t0_1_29 = rt0_1_29 * (4 + t0_0_59);
t0_1_29 += (1 - rt0_1_29) * t0_0_58;

int rt0_1_30 = (input[i] > (243+offset0));
int t0_1_30 = rt0_1_30 * (4 + t0_0_61);
t0_1_30 += (1 - rt0_1_30) * t0_0_60;

int rt0_1_31 = (input[i] > (251+offset0));
int t0_1_31 = rt0_1_31 * (4 + t0_0_63);
t0_1_31 += (1 - rt0_1_31) * t0_0_62;

int rt0_2_0 = (input[i] > (7+offset0));
int t0_2_0 = rt0_2_0 * (8 + t0_1_1);
t0_2_0 += (1 - rt0_2_0) * t0_1_0;

int rt0_2_1 = (input[i] > (23+offset0));
int t0_2_1 = rt0_2_1 * (8 + t0_1_3);
t0_2_1 += (1 - rt0_2_1) * t0_1_2;

int rt0_2_2 = (input[i] > (39+offset0));
int t0_2_2 = rt0_2_2 * (8 + t0_1_5);
t0_2_2 += (1 - rt0_2_2) * t0_1_4;

int rt0_2_3 = (input[i] > (55+offset0));
int t0_2_3 = rt0_2_3 * (8 + t0_1_7);
t0_2_3 += (1 - rt0_2_3) * t0_1_6;

int rt0_2_4 = (input[i] > (71+offset0));
int t0_2_4 = rt0_2_4 * (8 + t0_1_9);
t0_2_4 += (1 - rt0_2_4) * t0_1_8;

int rt0_2_5 = (input[i] > (87+offset0));
int t0_2_5 = rt0_2_5 * (8 + t0_1_11);
t0_2_5 += (1 - rt0_2_5) * t0_1_10;

int rt0_2_6 = (input[i] > (103+offset0));
int t0_2_6 = rt0_2_6 * (8 + t0_1_13);
t0_2_6 += (1 - rt0_2_6) * t0_1_12;

int rt0_2_7 = (input[i] > (119+offset0));
int t0_2_7 = rt0_2_7 * (8 + t0_1_15);
t0_2_7 += (1 - rt0_2_7) * t0_1_14;

int rt0_2_8 = (input[i] > (135+offset0));
int t0_2_8 = rt0_2_8 * (8 + t0_1_17);
t0_2_8 += (1 - rt0_2_8) * t0_1_16;

int rt0_2_9 = (input[i] > (151+offset0));
int t0_2_9 = rt0_2_9 * (8 + t0_1_19);
t0_2_9 += (1 - rt0_2_9) * t0_1_18;

int rt0_2_10 = (input[i] > (167+offset0));
int t0_2_10 = rt0_2_10 * (8 + t0_1_21);
t0_2_10 += (1 - rt0_2_10) * t0_1_20;

int rt0_2_11 = (input[i] > (183+offset0));
int t0_2_11 = rt0_2_11 * (8 + t0_1_23);
t0_2_11 += (1 - rt0_2_11) * t0_1_22;

int rt0_2_12 = (input[i] > (199+offset0));
int t0_2_12 = rt0_2_12 * (8 + t0_1_25);
t0_2_12 += (1 - rt0_2_12) * t0_1_24;

int rt0_2_13 = (input[i] > (215+offset0));
int t0_2_13 = rt0_2_13 * (8 + t0_1_27);
t0_2_13 += (1 - rt0_2_13) * t0_1_26;

int rt0_2_14 = (input[i] > (231+offset0));
int t0_2_14 = rt0_2_14 * (8 + t0_1_29);
t0_2_14 += (1 - rt0_2_14) * t0_1_28;

int rt0_2_15 = (input[i] > (247+offset0));
int t0_2_15 = rt0_2_15 * (8 + t0_1_31);
t0_2_15 += (1 - rt0_2_15) * t0_1_30;

int rt0_3_0 = (input[i] > (15+offset0));
int t0_3_0 = rt0_3_0 * (16 + t0_2_1);
t0_3_0 += (1 - rt0_3_0) * t0_2_0;

int rt0_3_1 = (input[i] > (47+offset0));
int t0_3_1 = rt0_3_1 * (16 + t0_2_3);
t0_3_1 += (1 - rt0_3_1) * t0_2_2;

int rt0_3_2 = (input[i] > (79+offset0));
int t0_3_2 = rt0_3_2 * (16 + t0_2_5);
t0_3_2 += (1 - rt0_3_2) * t0_2_4;

int rt0_3_3 = (input[i] > (111+offset0));
int t0_3_3 = rt0_3_3 * (16 + t0_2_7);
t0_3_3 += (1 - rt0_3_3) * t0_2_6;

int rt0_3_4 = (input[i] > (143+offset0));
int t0_3_4 = rt0_3_4 * (16 + t0_2_9);
t0_3_4 += (1 - rt0_3_4) * t0_2_8;

int rt0_3_5 = (input[i] > (175+offset0));
int t0_3_5 = rt0_3_5 * (16 + t0_2_11);
t0_3_5 += (1 - rt0_3_5) * t0_2_10;

int rt0_3_6 = (input[i] > (207+offset0));
int t0_3_6 = rt0_3_6 * (16 + t0_2_13);
t0_3_6 += (1 - rt0_3_6) * t0_2_12;

int rt0_3_7 = (input[i] > (239+offset0));
int t0_3_7 = rt0_3_7 * (16 + t0_2_15);
t0_3_7 += (1 - rt0_3_7) * t0_2_14;

int rt0_4_0 = (input[i] > (31+offset0));
int t0_4_0 = rt0_4_0 * (32 + t0_3_1);
t0_4_0 += (1 - rt0_4_0) * t0_3_0;

int rt0_4_1 = (input[i] > (95+offset0));
int t0_4_1 = rt0_4_1 * (32 + t0_3_3);
t0_4_1 += (1 - rt0_4_1) * t0_3_2;

int rt0_4_2 = (input[i] > (159+offset0));
int t0_4_2 = rt0_4_2 * (32 + t0_3_5);
t0_4_2 += (1 - rt0_4_2) * t0_3_4;

int rt0_4_3 = (input[i] > (223+offset0));
int t0_4_3 = rt0_4_3 * (32 + t0_3_7);
t0_4_3 += (1 - rt0_4_3) * t0_3_6;

int rt0_5_0 = (input[i] > (63+offset0));
int t0_5_0 = rt0_5_0 * (64 + t0_4_1);
t0_5_0 += (1 - rt0_5_0) * t0_4_0;

int rt0_5_1 = (input[i] > (191+offset0));
int t0_5_1 = rt0_5_1 * (64 + t0_4_3);
t0_5_1 += (1 - rt0_5_1) * t0_4_2;

int rt0_6_0 = (input[i] > (127+offset0));
int t0_6_0 = rt0_6_0 * (128 + t0_5_1);
t0_6_0 += (1 - rt0_6_0) * t0_5_0;

vR[i] = t0_6_0;
	}
}

int main(int argc, char **argv)
{
	float elapsed_time;
	// set up device
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("[CUDA - Random Forest]: EXP %s\n", EXP_NAME);
	printf("[CUDA - Random Forest]: Using Device %d: %s\n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));

	// set up data size of vectors
	int nElem = N_ELEM;
	printf("[CUDA - Random Forest]: Vector Size %d\n", nElem);

	// malloc host memory
	size_t nBytes = nElem * sizeof(float);
	float *h_vR;
	h_vR = (float *)malloc(nBytes);
	float *h_input;
	h_input = (float *)malloc(nBytes);

	printf("[CUDA - Random Forest]: Start Reading Dataset\n");
	initData(h_input, nElem);
	printf("[CUDA - Random Forest]: Finished Reading Dataset\n");
	memset(h_vR, 0, nBytes);

	// malloc device global memory
	float *d_vR;
	CHECK(hipMalloc((float **)&d_vR, nBytes));
	float *d_input;
	CHECK(hipMalloc((float **)&d_input, nBytes));

	// transfer data from host to device
	CHECK(hipMemcpy(d_input, h_input, nBytes, hipMemcpyHostToDevice));

	// invoke kernel at host side
	int iLen = 512;
	dim3 block(iLen);
	dim3 grid((nElem + block.x - 1) / block.x);
	hipEvent_t start, stop;
	CHECK(hipEventCreate(&start));
	CHECK(hipEventCreate(&stop));

	// record start event
	CHECK(hipEventRecord(start, 0));
#if defined(_2TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, d_vR);
#elif defined(_3TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, d_vR);
#elif defined(_4TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, d_vR);
#else
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, d_vR);
#endif
	CHECK(hipEventRecord(stop, 0));
	CHECK(hipEventSynchronize(stop));
	hipDeviceSynchronize();

	// calculate elapsed time
	CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
	printf("[CUDA - Random Forest]: %s - execution time = %.6fms\n", EXP_NAME, elapsed_time);
	registerTime(elapsed_time);

	// copy kernel result back to host side
	CHECK(hipMemcpy(h_vR, d_vR, nBytes, hipMemcpyDeviceToHost));
	CHECK(hipGetLastError());

	//Verify the answers
	char flag = 1;
	for (int i = 0; i < nElem; i++)
	{
		if (i < 32)
		{
			printf("%.3f ", h_vR[i]);
		}
		if (h_input[i] != h_vR[i])
		{
			flag = 0;
			break;
		}
	}
	if (flag == 1)
	{
//		printf("Return correct");
	}
	else
	{
//		printf("Error!!");
	}
	printf("\n ");

	// free device memory
	hipFree(d_input);
	hipFree(d_vR);

	// free host memory
	free(h_input);
	free(h_vR);
	return (0);
}

void initData(float *ip, int size)
{
	for (int i = 0; i < size; i++)
	{
		ip[i] = (float)(0);
	}
	return;
}

void registerTime(float value)
{
	printf("[CUDA - Random Forest]: Time spent: %f.\n", value);
}
