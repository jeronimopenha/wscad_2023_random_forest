#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>

#define N_ELEM 50000000

#define WARP_SIZE 32

#define CHECK(call)                                                \
	{                                                              \
		const hipError_t error = call;                            \
		if (error != hipSuccess)                                  \
		{                                                          \
			fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
			fprintf(stderr, "code: %d, reason: %s\n", error,       \
					hipGetErrorString(error));                    \
		}                                                          \
	}

void initData(float *ip, int size);
void registerTime(float value);

#define _MOD 256
#define _4TREE
#define EXP_NAME "_RF_CALC_4TREE_8LV_1INPUT_50000000"

// RF_with_if

__global__ void RF_kernel(float *input, const int n, const int offset0, const int offset1, const int offset2, const int offset3, float *vR)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
int rt0_0_0 = (input[i] > (1+offset0));
int t0_0_0 = rt0_0_0 * (2 + (input[i] > (2+offset0)));
t0_0_0 += (1 - rt0_0_0) * (input[i] > (0+offset0));

int rt0_0_1 = (input[i] > (5+offset0));
int t0_0_1 = rt0_0_1 * (2 + (input[i] > (6+offset0)));
t0_0_1 += (1 - rt0_0_1) * (input[i] > (4+offset0));

int rt0_0_2 = (input[i] > (9+offset0));
int t0_0_2 = rt0_0_2 * (2 + (input[i] > (10+offset0)));
t0_0_2 += (1 - rt0_0_2) * (input[i] > (8+offset0));

int rt0_0_3 = (input[i] > (13+offset0));
int t0_0_3 = rt0_0_3 * (2 + (input[i] > (14+offset0)));
t0_0_3 += (1 - rt0_0_3) * (input[i] > (12+offset0));

int rt0_0_4 = (input[i] > (17+offset0));
int t0_0_4 = rt0_0_4 * (2 + (input[i] > (18+offset0)));
t0_0_4 += (1 - rt0_0_4) * (input[i] > (16+offset0));

int rt0_0_5 = (input[i] > (21+offset0));
int t0_0_5 = rt0_0_5 * (2 + (input[i] > (22+offset0)));
t0_0_5 += (1 - rt0_0_5) * (input[i] > (20+offset0));

int rt0_0_6 = (input[i] > (25+offset0));
int t0_0_6 = rt0_0_6 * (2 + (input[i] > (26+offset0)));
t0_0_6 += (1 - rt0_0_6) * (input[i] > (24+offset0));

int rt0_0_7 = (input[i] > (29+offset0));
int t0_0_7 = rt0_0_7 * (2 + (input[i] > (30+offset0)));
t0_0_7 += (1 - rt0_0_7) * (input[i] > (28+offset0));

int rt0_0_8 = (input[i] > (33+offset0));
int t0_0_8 = rt0_0_8 * (2 + (input[i] > (34+offset0)));
t0_0_8 += (1 - rt0_0_8) * (input[i] > (32+offset0));

int rt0_0_9 = (input[i] > (37+offset0));
int t0_0_9 = rt0_0_9 * (2 + (input[i] > (38+offset0)));
t0_0_9 += (1 - rt0_0_9) * (input[i] > (36+offset0));

int rt0_0_10 = (input[i] > (41+offset0));
int t0_0_10 = rt0_0_10 * (2 + (input[i] > (42+offset0)));
t0_0_10 += (1 - rt0_0_10) * (input[i] > (40+offset0));

int rt0_0_11 = (input[i] > (45+offset0));
int t0_0_11 = rt0_0_11 * (2 + (input[i] > (46+offset0)));
t0_0_11 += (1 - rt0_0_11) * (input[i] > (44+offset0));

int rt0_0_12 = (input[i] > (49+offset0));
int t0_0_12 = rt0_0_12 * (2 + (input[i] > (50+offset0)));
t0_0_12 += (1 - rt0_0_12) * (input[i] > (48+offset0));

int rt0_0_13 = (input[i] > (53+offset0));
int t0_0_13 = rt0_0_13 * (2 + (input[i] > (54+offset0)));
t0_0_13 += (1 - rt0_0_13) * (input[i] > (52+offset0));

int rt0_0_14 = (input[i] > (57+offset0));
int t0_0_14 = rt0_0_14 * (2 + (input[i] > (58+offset0)));
t0_0_14 += (1 - rt0_0_14) * (input[i] > (56+offset0));

int rt0_0_15 = (input[i] > (61+offset0));
int t0_0_15 = rt0_0_15 * (2 + (input[i] > (62+offset0)));
t0_0_15 += (1 - rt0_0_15) * (input[i] > (60+offset0));

int rt0_0_16 = (input[i] > (65+offset0));
int t0_0_16 = rt0_0_16 * (2 + (input[i] > (66+offset0)));
t0_0_16 += (1 - rt0_0_16) * (input[i] > (64+offset0));

int rt0_0_17 = (input[i] > (69+offset0));
int t0_0_17 = rt0_0_17 * (2 + (input[i] > (70+offset0)));
t0_0_17 += (1 - rt0_0_17) * (input[i] > (68+offset0));

int rt0_0_18 = (input[i] > (73+offset0));
int t0_0_18 = rt0_0_18 * (2 + (input[i] > (74+offset0)));
t0_0_18 += (1 - rt0_0_18) * (input[i] > (72+offset0));

int rt0_0_19 = (input[i] > (77+offset0));
int t0_0_19 = rt0_0_19 * (2 + (input[i] > (78+offset0)));
t0_0_19 += (1 - rt0_0_19) * (input[i] > (76+offset0));

int rt0_0_20 = (input[i] > (81+offset0));
int t0_0_20 = rt0_0_20 * (2 + (input[i] > (82+offset0)));
t0_0_20 += (1 - rt0_0_20) * (input[i] > (80+offset0));

int rt0_0_21 = (input[i] > (85+offset0));
int t0_0_21 = rt0_0_21 * (2 + (input[i] > (86+offset0)));
t0_0_21 += (1 - rt0_0_21) * (input[i] > (84+offset0));

int rt0_0_22 = (input[i] > (89+offset0));
int t0_0_22 = rt0_0_22 * (2 + (input[i] > (90+offset0)));
t0_0_22 += (1 - rt0_0_22) * (input[i] > (88+offset0));

int rt0_0_23 = (input[i] > (93+offset0));
int t0_0_23 = rt0_0_23 * (2 + (input[i] > (94+offset0)));
t0_0_23 += (1 - rt0_0_23) * (input[i] > (92+offset0));

int rt0_0_24 = (input[i] > (97+offset0));
int t0_0_24 = rt0_0_24 * (2 + (input[i] > (98+offset0)));
t0_0_24 += (1 - rt0_0_24) * (input[i] > (96+offset0));

int rt0_0_25 = (input[i] > (101+offset0));
int t0_0_25 = rt0_0_25 * (2 + (input[i] > (102+offset0)));
t0_0_25 += (1 - rt0_0_25) * (input[i] > (100+offset0));

int rt0_0_26 = (input[i] > (105+offset0));
int t0_0_26 = rt0_0_26 * (2 + (input[i] > (106+offset0)));
t0_0_26 += (1 - rt0_0_26) * (input[i] > (104+offset0));

int rt0_0_27 = (input[i] > (109+offset0));
int t0_0_27 = rt0_0_27 * (2 + (input[i] > (110+offset0)));
t0_0_27 += (1 - rt0_0_27) * (input[i] > (108+offset0));

int rt0_0_28 = (input[i] > (113+offset0));
int t0_0_28 = rt0_0_28 * (2 + (input[i] > (114+offset0)));
t0_0_28 += (1 - rt0_0_28) * (input[i] > (112+offset0));

int rt0_0_29 = (input[i] > (117+offset0));
int t0_0_29 = rt0_0_29 * (2 + (input[i] > (118+offset0)));
t0_0_29 += (1 - rt0_0_29) * (input[i] > (116+offset0));

int rt0_0_30 = (input[i] > (121+offset0));
int t0_0_30 = rt0_0_30 * (2 + (input[i] > (122+offset0)));
t0_0_30 += (1 - rt0_0_30) * (input[i] > (120+offset0));

int rt0_0_31 = (input[i] > (125+offset0));
int t0_0_31 = rt0_0_31 * (2 + (input[i] > (126+offset0)));
t0_0_31 += (1 - rt0_0_31) * (input[i] > (124+offset0));

int rt0_0_32 = (input[i] > (129+offset0));
int t0_0_32 = rt0_0_32 * (2 + (input[i] > (130+offset0)));
t0_0_32 += (1 - rt0_0_32) * (input[i] > (128+offset0));

int rt0_0_33 = (input[i] > (133+offset0));
int t0_0_33 = rt0_0_33 * (2 + (input[i] > (134+offset0)));
t0_0_33 += (1 - rt0_0_33) * (input[i] > (132+offset0));

int rt0_0_34 = (input[i] > (137+offset0));
int t0_0_34 = rt0_0_34 * (2 + (input[i] > (138+offset0)));
t0_0_34 += (1 - rt0_0_34) * (input[i] > (136+offset0));

int rt0_0_35 = (input[i] > (141+offset0));
int t0_0_35 = rt0_0_35 * (2 + (input[i] > (142+offset0)));
t0_0_35 += (1 - rt0_0_35) * (input[i] > (140+offset0));

int rt0_0_36 = (input[i] > (145+offset0));
int t0_0_36 = rt0_0_36 * (2 + (input[i] > (146+offset0)));
t0_0_36 += (1 - rt0_0_36) * (input[i] > (144+offset0));

int rt0_0_37 = (input[i] > (149+offset0));
int t0_0_37 = rt0_0_37 * (2 + (input[i] > (150+offset0)));
t0_0_37 += (1 - rt0_0_37) * (input[i] > (148+offset0));

int rt0_0_38 = (input[i] > (153+offset0));
int t0_0_38 = rt0_0_38 * (2 + (input[i] > (154+offset0)));
t0_0_38 += (1 - rt0_0_38) * (input[i] > (152+offset0));

int rt0_0_39 = (input[i] > (157+offset0));
int t0_0_39 = rt0_0_39 * (2 + (input[i] > (158+offset0)));
t0_0_39 += (1 - rt0_0_39) * (input[i] > (156+offset0));

int rt0_0_40 = (input[i] > (161+offset0));
int t0_0_40 = rt0_0_40 * (2 + (input[i] > (162+offset0)));
t0_0_40 += (1 - rt0_0_40) * (input[i] > (160+offset0));

int rt0_0_41 = (input[i] > (165+offset0));
int t0_0_41 = rt0_0_41 * (2 + (input[i] > (166+offset0)));
t0_0_41 += (1 - rt0_0_41) * (input[i] > (164+offset0));

int rt0_0_42 = (input[i] > (169+offset0));
int t0_0_42 = rt0_0_42 * (2 + (input[i] > (170+offset0)));
t0_0_42 += (1 - rt0_0_42) * (input[i] > (168+offset0));

int rt0_0_43 = (input[i] > (173+offset0));
int t0_0_43 = rt0_0_43 * (2 + (input[i] > (174+offset0)));
t0_0_43 += (1 - rt0_0_43) * (input[i] > (172+offset0));

int rt0_0_44 = (input[i] > (177+offset0));
int t0_0_44 = rt0_0_44 * (2 + (input[i] > (178+offset0)));
t0_0_44 += (1 - rt0_0_44) * (input[i] > (176+offset0));

int rt0_0_45 = (input[i] > (181+offset0));
int t0_0_45 = rt0_0_45 * (2 + (input[i] > (182+offset0)));
t0_0_45 += (1 - rt0_0_45) * (input[i] > (180+offset0));

int rt0_0_46 = (input[i] > (185+offset0));
int t0_0_46 = rt0_0_46 * (2 + (input[i] > (186+offset0)));
t0_0_46 += (1 - rt0_0_46) * (input[i] > (184+offset0));

int rt0_0_47 = (input[i] > (189+offset0));
int t0_0_47 = rt0_0_47 * (2 + (input[i] > (190+offset0)));
t0_0_47 += (1 - rt0_0_47) * (input[i] > (188+offset0));

int rt0_0_48 = (input[i] > (193+offset0));
int t0_0_48 = rt0_0_48 * (2 + (input[i] > (194+offset0)));
t0_0_48 += (1 - rt0_0_48) * (input[i] > (192+offset0));

int rt0_0_49 = (input[i] > (197+offset0));
int t0_0_49 = rt0_0_49 * (2 + (input[i] > (198+offset0)));
t0_0_49 += (1 - rt0_0_49) * (input[i] > (196+offset0));

int rt0_0_50 = (input[i] > (201+offset0));
int t0_0_50 = rt0_0_50 * (2 + (input[i] > (202+offset0)));
t0_0_50 += (1 - rt0_0_50) * (input[i] > (200+offset0));

int rt0_0_51 = (input[i] > (205+offset0));
int t0_0_51 = rt0_0_51 * (2 + (input[i] > (206+offset0)));
t0_0_51 += (1 - rt0_0_51) * (input[i] > (204+offset0));

int rt0_0_52 = (input[i] > (209+offset0));
int t0_0_52 = rt0_0_52 * (2 + (input[i] > (210+offset0)));
t0_0_52 += (1 - rt0_0_52) * (input[i] > (208+offset0));

int rt0_0_53 = (input[i] > (213+offset0));
int t0_0_53 = rt0_0_53 * (2 + (input[i] > (214+offset0)));
t0_0_53 += (1 - rt0_0_53) * (input[i] > (212+offset0));

int rt0_0_54 = (input[i] > (217+offset0));
int t0_0_54 = rt0_0_54 * (2 + (input[i] > (218+offset0)));
t0_0_54 += (1 - rt0_0_54) * (input[i] > (216+offset0));

int rt0_0_55 = (input[i] > (221+offset0));
int t0_0_55 = rt0_0_55 * (2 + (input[i] > (222+offset0)));
t0_0_55 += (1 - rt0_0_55) * (input[i] > (220+offset0));

int rt0_0_56 = (input[i] > (225+offset0));
int t0_0_56 = rt0_0_56 * (2 + (input[i] > (226+offset0)));
t0_0_56 += (1 - rt0_0_56) * (input[i] > (224+offset0));

int rt0_0_57 = (input[i] > (229+offset0));
int t0_0_57 = rt0_0_57 * (2 + (input[i] > (230+offset0)));
t0_0_57 += (1 - rt0_0_57) * (input[i] > (228+offset0));

int rt0_0_58 = (input[i] > (233+offset0));
int t0_0_58 = rt0_0_58 * (2 + (input[i] > (234+offset0)));
t0_0_58 += (1 - rt0_0_58) * (input[i] > (232+offset0));

int rt0_0_59 = (input[i] > (237+offset0));
int t0_0_59 = rt0_0_59 * (2 + (input[i] > (238+offset0)));
t0_0_59 += (1 - rt0_0_59) * (input[i] > (236+offset0));

int rt0_0_60 = (input[i] > (241+offset0));
int t0_0_60 = rt0_0_60 * (2 + (input[i] > (242+offset0)));
t0_0_60 += (1 - rt0_0_60) * (input[i] > (240+offset0));

int rt0_0_61 = (input[i] > (245+offset0));
int t0_0_61 = rt0_0_61 * (2 + (input[i] > (246+offset0)));
t0_0_61 += (1 - rt0_0_61) * (input[i] > (244+offset0));

int rt0_0_62 = (input[i] > (249+offset0));
int t0_0_62 = rt0_0_62 * (2 + (input[i] > (250+offset0)));
t0_0_62 += (1 - rt0_0_62) * (input[i] > (248+offset0));

int rt0_0_63 = (input[i] > (253+offset0));
int t0_0_63 = rt0_0_63 * (2 + (input[i] > (254+offset0)));
t0_0_63 += (1 - rt0_0_63) * (input[i] > (252+offset0));

int rt0_1_0 = (input[i] > (3+offset0));
int t0_1_0 = rt0_1_0 * (4 + t0_0_1);
t0_1_0 += (1 - rt0_1_0) * t0_0_0;

int rt0_1_1 = (input[i] > (11+offset0));
int t0_1_1 = rt0_1_1 * (4 + t0_0_3);
t0_1_1 += (1 - rt0_1_1) * t0_0_2;

int rt0_1_2 = (input[i] > (19+offset0));
int t0_1_2 = rt0_1_2 * (4 + t0_0_5);
t0_1_2 += (1 - rt0_1_2) * t0_0_4;

int rt0_1_3 = (input[i] > (27+offset0));
int t0_1_3 = rt0_1_3 * (4 + t0_0_7);
t0_1_3 += (1 - rt0_1_3) * t0_0_6;

int rt0_1_4 = (input[i] > (35+offset0));
int t0_1_4 = rt0_1_4 * (4 + t0_0_9);
t0_1_4 += (1 - rt0_1_4) * t0_0_8;

int rt0_1_5 = (input[i] > (43+offset0));
int t0_1_5 = rt0_1_5 * (4 + t0_0_11);
t0_1_5 += (1 - rt0_1_5) * t0_0_10;

int rt0_1_6 = (input[i] > (51+offset0));
int t0_1_6 = rt0_1_6 * (4 + t0_0_13);
t0_1_6 += (1 - rt0_1_6) * t0_0_12;

int rt0_1_7 = (input[i] > (59+offset0));
int t0_1_7 = rt0_1_7 * (4 + t0_0_15);
t0_1_7 += (1 - rt0_1_7) * t0_0_14;

int rt0_1_8 = (input[i] > (67+offset0));
int t0_1_8 = rt0_1_8 * (4 + t0_0_17);
t0_1_8 += (1 - rt0_1_8) * t0_0_16;

int rt0_1_9 = (input[i] > (75+offset0));
int t0_1_9 = rt0_1_9 * (4 + t0_0_19);
t0_1_9 += (1 - rt0_1_9) * t0_0_18;

int rt0_1_10 = (input[i] > (83+offset0));
int t0_1_10 = rt0_1_10 * (4 + t0_0_21);
t0_1_10 += (1 - rt0_1_10) * t0_0_20;

int rt0_1_11 = (input[i] > (91+offset0));
int t0_1_11 = rt0_1_11 * (4 + t0_0_23);
t0_1_11 += (1 - rt0_1_11) * t0_0_22;

int rt0_1_12 = (input[i] > (99+offset0));
int t0_1_12 = rt0_1_12 * (4 + t0_0_25);
t0_1_12 += (1 - rt0_1_12) * t0_0_24;

int rt0_1_13 = (input[i] > (107+offset0));
int t0_1_13 = rt0_1_13 * (4 + t0_0_27);
t0_1_13 += (1 - rt0_1_13) * t0_0_26;

int rt0_1_14 = (input[i] > (115+offset0));
int t0_1_14 = rt0_1_14 * (4 + t0_0_29);
t0_1_14 += (1 - rt0_1_14) * t0_0_28;

int rt0_1_15 = (input[i] > (123+offset0));
int t0_1_15 = rt0_1_15 * (4 + t0_0_31);
t0_1_15 += (1 - rt0_1_15) * t0_0_30;

int rt0_1_16 = (input[i] > (131+offset0));
int t0_1_16 = rt0_1_16 * (4 + t0_0_33);
t0_1_16 += (1 - rt0_1_16) * t0_0_32;

int rt0_1_17 = (input[i] > (139+offset0));
int t0_1_17 = rt0_1_17 * (4 + t0_0_35);
t0_1_17 += (1 - rt0_1_17) * t0_0_34;

int rt0_1_18 = (input[i] > (147+offset0));
int t0_1_18 = rt0_1_18 * (4 + t0_0_37);
t0_1_18 += (1 - rt0_1_18) * t0_0_36;

int rt0_1_19 = (input[i] > (155+offset0));
int t0_1_19 = rt0_1_19 * (4 + t0_0_39);
t0_1_19 += (1 - rt0_1_19) * t0_0_38;

int rt0_1_20 = (input[i] > (163+offset0));
int t0_1_20 = rt0_1_20 * (4 + t0_0_41);
t0_1_20 += (1 - rt0_1_20) * t0_0_40;

int rt0_1_21 = (input[i] > (171+offset0));
int t0_1_21 = rt0_1_21 * (4 + t0_0_43);
t0_1_21 += (1 - rt0_1_21) * t0_0_42;

int rt0_1_22 = (input[i] > (179+offset0));
int t0_1_22 = rt0_1_22 * (4 + t0_0_45);
t0_1_22 += (1 - rt0_1_22) * t0_0_44;

int rt0_1_23 = (input[i] > (187+offset0));
int t0_1_23 = rt0_1_23 * (4 + t0_0_47);
t0_1_23 += (1 - rt0_1_23) * t0_0_46;

int rt0_1_24 = (input[i] > (195+offset0));
int t0_1_24 = rt0_1_24 * (4 + t0_0_49);
t0_1_24 += (1 - rt0_1_24) * t0_0_48;

int rt0_1_25 = (input[i] > (203+offset0));
int t0_1_25 = rt0_1_25 * (4 + t0_0_51);
t0_1_25 += (1 - rt0_1_25) * t0_0_50;

int rt0_1_26 = (input[i] > (211+offset0));
int t0_1_26 = rt0_1_26 * (4 + t0_0_53);
t0_1_26 += (1 - rt0_1_26) * t0_0_52;

int rt0_1_27 = (input[i] > (219+offset0));
int t0_1_27 = rt0_1_27 * (4 + t0_0_55);
t0_1_27 += (1 - rt0_1_27) * t0_0_54;

int rt0_1_28 = (input[i] > (227+offset0));
int t0_1_28 = rt0_1_28 * (4 + t0_0_57);
t0_1_28 += (1 - rt0_1_28) * t0_0_56;

int rt0_1_29 = (input[i] > (235+offset0));
int t0_1_29 = rt0_1_29 * (4 + t0_0_59);
t0_1_29 += (1 - rt0_1_29) * t0_0_58;

int rt0_1_30 = (input[i] > (243+offset0));
int t0_1_30 = rt0_1_30 * (4 + t0_0_61);
t0_1_30 += (1 - rt0_1_30) * t0_0_60;

int rt0_1_31 = (input[i] > (251+offset0));
int t0_1_31 = rt0_1_31 * (4 + t0_0_63);
t0_1_31 += (1 - rt0_1_31) * t0_0_62;

int rt0_2_0 = (input[i] > (7+offset0));
int t0_2_0 = rt0_2_0 * (8 + t0_1_1);
t0_2_0 += (1 - rt0_2_0) * t0_1_0;

int rt0_2_1 = (input[i] > (23+offset0));
int t0_2_1 = rt0_2_1 * (8 + t0_1_3);
t0_2_1 += (1 - rt0_2_1) * t0_1_2;

int rt0_2_2 = (input[i] > (39+offset0));
int t0_2_2 = rt0_2_2 * (8 + t0_1_5);
t0_2_2 += (1 - rt0_2_2) * t0_1_4;

int rt0_2_3 = (input[i] > (55+offset0));
int t0_2_3 = rt0_2_3 * (8 + t0_1_7);
t0_2_3 += (1 - rt0_2_3) * t0_1_6;

int rt0_2_4 = (input[i] > (71+offset0));
int t0_2_4 = rt0_2_4 * (8 + t0_1_9);
t0_2_4 += (1 - rt0_2_4) * t0_1_8;

int rt0_2_5 = (input[i] > (87+offset0));
int t0_2_5 = rt0_2_5 * (8 + t0_1_11);
t0_2_5 += (1 - rt0_2_5) * t0_1_10;

int rt0_2_6 = (input[i] > (103+offset0));
int t0_2_6 = rt0_2_6 * (8 + t0_1_13);
t0_2_6 += (1 - rt0_2_6) * t0_1_12;

int rt0_2_7 = (input[i] > (119+offset0));
int t0_2_7 = rt0_2_7 * (8 + t0_1_15);
t0_2_7 += (1 - rt0_2_7) * t0_1_14;

int rt0_2_8 = (input[i] > (135+offset0));
int t0_2_8 = rt0_2_8 * (8 + t0_1_17);
t0_2_8 += (1 - rt0_2_8) * t0_1_16;

int rt0_2_9 = (input[i] > (151+offset0));
int t0_2_9 = rt0_2_9 * (8 + t0_1_19);
t0_2_9 += (1 - rt0_2_9) * t0_1_18;

int rt0_2_10 = (input[i] > (167+offset0));
int t0_2_10 = rt0_2_10 * (8 + t0_1_21);
t0_2_10 += (1 - rt0_2_10) * t0_1_20;

int rt0_2_11 = (input[i] > (183+offset0));
int t0_2_11 = rt0_2_11 * (8 + t0_1_23);
t0_2_11 += (1 - rt0_2_11) * t0_1_22;

int rt0_2_12 = (input[i] > (199+offset0));
int t0_2_12 = rt0_2_12 * (8 + t0_1_25);
t0_2_12 += (1 - rt0_2_12) * t0_1_24;

int rt0_2_13 = (input[i] > (215+offset0));
int t0_2_13 = rt0_2_13 * (8 + t0_1_27);
t0_2_13 += (1 - rt0_2_13) * t0_1_26;

int rt0_2_14 = (input[i] > (231+offset0));
int t0_2_14 = rt0_2_14 * (8 + t0_1_29);
t0_2_14 += (1 - rt0_2_14) * t0_1_28;

int rt0_2_15 = (input[i] > (247+offset0));
int t0_2_15 = rt0_2_15 * (8 + t0_1_31);
t0_2_15 += (1 - rt0_2_15) * t0_1_30;

int rt0_3_0 = (input[i] > (15+offset0));
int t0_3_0 = rt0_3_0 * (16 + t0_2_1);
t0_3_0 += (1 - rt0_3_0) * t0_2_0;

int rt0_3_1 = (input[i] > (47+offset0));
int t0_3_1 = rt0_3_1 * (16 + t0_2_3);
t0_3_1 += (1 - rt0_3_1) * t0_2_2;

int rt0_3_2 = (input[i] > (79+offset0));
int t0_3_2 = rt0_3_2 * (16 + t0_2_5);
t0_3_2 += (1 - rt0_3_2) * t0_2_4;

int rt0_3_3 = (input[i] > (111+offset0));
int t0_3_3 = rt0_3_3 * (16 + t0_2_7);
t0_3_3 += (1 - rt0_3_3) * t0_2_6;

int rt0_3_4 = (input[i] > (143+offset0));
int t0_3_4 = rt0_3_4 * (16 + t0_2_9);
t0_3_4 += (1 - rt0_3_4) * t0_2_8;

int rt0_3_5 = (input[i] > (175+offset0));
int t0_3_5 = rt0_3_5 * (16 + t0_2_11);
t0_3_5 += (1 - rt0_3_5) * t0_2_10;

int rt0_3_6 = (input[i] > (207+offset0));
int t0_3_6 = rt0_3_6 * (16 + t0_2_13);
t0_3_6 += (1 - rt0_3_6) * t0_2_12;

int rt0_3_7 = (input[i] > (239+offset0));
int t0_3_7 = rt0_3_7 * (16 + t0_2_15);
t0_3_7 += (1 - rt0_3_7) * t0_2_14;

int rt0_4_0 = (input[i] > (31+offset0));
int t0_4_0 = rt0_4_0 * (32 + t0_3_1);
t0_4_0 += (1 - rt0_4_0) * t0_3_0;

int rt0_4_1 = (input[i] > (95+offset0));
int t0_4_1 = rt0_4_1 * (32 + t0_3_3);
t0_4_1 += (1 - rt0_4_1) * t0_3_2;

int rt0_4_2 = (input[i] > (159+offset0));
int t0_4_2 = rt0_4_2 * (32 + t0_3_5);
t0_4_2 += (1 - rt0_4_2) * t0_3_4;

int rt0_4_3 = (input[i] > (223+offset0));
int t0_4_3 = rt0_4_3 * (32 + t0_3_7);
t0_4_3 += (1 - rt0_4_3) * t0_3_6;

int rt0_5_0 = (input[i] > (63+offset0));
int t0_5_0 = rt0_5_0 * (64 + t0_4_1);
t0_5_0 += (1 - rt0_5_0) * t0_4_0;

int rt0_5_1 = (input[i] > (191+offset0));
int t0_5_1 = rt0_5_1 * (64 + t0_4_3);
t0_5_1 += (1 - rt0_5_1) * t0_4_2;

int rt0_6_0 = (input[i] > (127+offset0));
int t0_6_0 = rt0_6_0 * (128 + t0_5_1);
t0_6_0 += (1 - rt0_6_0) * t0_5_0;

int rt1_0_0 = (input[i] > (1+offset1));
int t1_0_0 = rt1_0_0 * (2 + (input[i] > (2+offset1)));
t1_0_0 += (1 - rt1_0_0) * (input[i] > (0+offset1));

int rt1_0_1 = (input[i] > (5+offset1));
int t1_0_1 = rt1_0_1 * (2 + (input[i] > (6+offset1)));
t1_0_1 += (1 - rt1_0_1) * (input[i] > (4+offset1));

int rt1_0_2 = (input[i] > (9+offset1));
int t1_0_2 = rt1_0_2 * (2 + (input[i] > (10+offset1)));
t1_0_2 += (1 - rt1_0_2) * (input[i] > (8+offset1));

int rt1_0_3 = (input[i] > (13+offset1));
int t1_0_3 = rt1_0_3 * (2 + (input[i] > (14+offset1)));
t1_0_3 += (1 - rt1_0_3) * (input[i] > (12+offset1));

int rt1_0_4 = (input[i] > (17+offset1));
int t1_0_4 = rt1_0_4 * (2 + (input[i] > (18+offset1)));
t1_0_4 += (1 - rt1_0_4) * (input[i] > (16+offset1));

int rt1_0_5 = (input[i] > (21+offset1));
int t1_0_5 = rt1_0_5 * (2 + (input[i] > (22+offset1)));
t1_0_5 += (1 - rt1_0_5) * (input[i] > (20+offset1));

int rt1_0_6 = (input[i] > (25+offset1));
int t1_0_6 = rt1_0_6 * (2 + (input[i] > (26+offset1)));
t1_0_6 += (1 - rt1_0_6) * (input[i] > (24+offset1));

int rt1_0_7 = (input[i] > (29+offset1));
int t1_0_7 = rt1_0_7 * (2 + (input[i] > (30+offset1)));
t1_0_7 += (1 - rt1_0_7) * (input[i] > (28+offset1));

int rt1_0_8 = (input[i] > (33+offset1));
int t1_0_8 = rt1_0_8 * (2 + (input[i] > (34+offset1)));
t1_0_8 += (1 - rt1_0_8) * (input[i] > (32+offset1));

int rt1_0_9 = (input[i] > (37+offset1));
int t1_0_9 = rt1_0_9 * (2 + (input[i] > (38+offset1)));
t1_0_9 += (1 - rt1_0_9) * (input[i] > (36+offset1));

int rt1_0_10 = (input[i] > (41+offset1));
int t1_0_10 = rt1_0_10 * (2 + (input[i] > (42+offset1)));
t1_0_10 += (1 - rt1_0_10) * (input[i] > (40+offset1));

int rt1_0_11 = (input[i] > (45+offset1));
int t1_0_11 = rt1_0_11 * (2 + (input[i] > (46+offset1)));
t1_0_11 += (1 - rt1_0_11) * (input[i] > (44+offset1));

int rt1_0_12 = (input[i] > (49+offset1));
int t1_0_12 = rt1_0_12 * (2 + (input[i] > (50+offset1)));
t1_0_12 += (1 - rt1_0_12) * (input[i] > (48+offset1));

int rt1_0_13 = (input[i] > (53+offset1));
int t1_0_13 = rt1_0_13 * (2 + (input[i] > (54+offset1)));
t1_0_13 += (1 - rt1_0_13) * (input[i] > (52+offset1));

int rt1_0_14 = (input[i] > (57+offset1));
int t1_0_14 = rt1_0_14 * (2 + (input[i] > (58+offset1)));
t1_0_14 += (1 - rt1_0_14) * (input[i] > (56+offset1));

int rt1_0_15 = (input[i] > (61+offset1));
int t1_0_15 = rt1_0_15 * (2 + (input[i] > (62+offset1)));
t1_0_15 += (1 - rt1_0_15) * (input[i] > (60+offset1));

int rt1_0_16 = (input[i] > (65+offset1));
int t1_0_16 = rt1_0_16 * (2 + (input[i] > (66+offset1)));
t1_0_16 += (1 - rt1_0_16) * (input[i] > (64+offset1));

int rt1_0_17 = (input[i] > (69+offset1));
int t1_0_17 = rt1_0_17 * (2 + (input[i] > (70+offset1)));
t1_0_17 += (1 - rt1_0_17) * (input[i] > (68+offset1));

int rt1_0_18 = (input[i] > (73+offset1));
int t1_0_18 = rt1_0_18 * (2 + (input[i] > (74+offset1)));
t1_0_18 += (1 - rt1_0_18) * (input[i] > (72+offset1));

int rt1_0_19 = (input[i] > (77+offset1));
int t1_0_19 = rt1_0_19 * (2 + (input[i] > (78+offset1)));
t1_0_19 += (1 - rt1_0_19) * (input[i] > (76+offset1));

int rt1_0_20 = (input[i] > (81+offset1));
int t1_0_20 = rt1_0_20 * (2 + (input[i] > (82+offset1)));
t1_0_20 += (1 - rt1_0_20) * (input[i] > (80+offset1));

int rt1_0_21 = (input[i] > (85+offset1));
int t1_0_21 = rt1_0_21 * (2 + (input[i] > (86+offset1)));
t1_0_21 += (1 - rt1_0_21) * (input[i] > (84+offset1));

int rt1_0_22 = (input[i] > (89+offset1));
int t1_0_22 = rt1_0_22 * (2 + (input[i] > (90+offset1)));
t1_0_22 += (1 - rt1_0_22) * (input[i] > (88+offset1));

int rt1_0_23 = (input[i] > (93+offset1));
int t1_0_23 = rt1_0_23 * (2 + (input[i] > (94+offset1)));
t1_0_23 += (1 - rt1_0_23) * (input[i] > (92+offset1));

int rt1_0_24 = (input[i] > (97+offset1));
int t1_0_24 = rt1_0_24 * (2 + (input[i] > (98+offset1)));
t1_0_24 += (1 - rt1_0_24) * (input[i] > (96+offset1));

int rt1_0_25 = (input[i] > (101+offset1));
int t1_0_25 = rt1_0_25 * (2 + (input[i] > (102+offset1)));
t1_0_25 += (1 - rt1_0_25) * (input[i] > (100+offset1));

int rt1_0_26 = (input[i] > (105+offset1));
int t1_0_26 = rt1_0_26 * (2 + (input[i] > (106+offset1)));
t1_0_26 += (1 - rt1_0_26) * (input[i] > (104+offset1));

int rt1_0_27 = (input[i] > (109+offset1));
int t1_0_27 = rt1_0_27 * (2 + (input[i] > (110+offset1)));
t1_0_27 += (1 - rt1_0_27) * (input[i] > (108+offset1));

int rt1_0_28 = (input[i] > (113+offset1));
int t1_0_28 = rt1_0_28 * (2 + (input[i] > (114+offset1)));
t1_0_28 += (1 - rt1_0_28) * (input[i] > (112+offset1));

int rt1_0_29 = (input[i] > (117+offset1));
int t1_0_29 = rt1_0_29 * (2 + (input[i] > (118+offset1)));
t1_0_29 += (1 - rt1_0_29) * (input[i] > (116+offset1));

int rt1_0_30 = (input[i] > (121+offset1));
int t1_0_30 = rt1_0_30 * (2 + (input[i] > (122+offset1)));
t1_0_30 += (1 - rt1_0_30) * (input[i] > (120+offset1));

int rt1_0_31 = (input[i] > (125+offset1));
int t1_0_31 = rt1_0_31 * (2 + (input[i] > (126+offset1)));
t1_0_31 += (1 - rt1_0_31) * (input[i] > (124+offset1));

int rt1_0_32 = (input[i] > (129+offset1));
int t1_0_32 = rt1_0_32 * (2 + (input[i] > (130+offset1)));
t1_0_32 += (1 - rt1_0_32) * (input[i] > (128+offset1));

int rt1_0_33 = (input[i] > (133+offset1));
int t1_0_33 = rt1_0_33 * (2 + (input[i] > (134+offset1)));
t1_0_33 += (1 - rt1_0_33) * (input[i] > (132+offset1));

int rt1_0_34 = (input[i] > (137+offset1));
int t1_0_34 = rt1_0_34 * (2 + (input[i] > (138+offset1)));
t1_0_34 += (1 - rt1_0_34) * (input[i] > (136+offset1));

int rt1_0_35 = (input[i] > (141+offset1));
int t1_0_35 = rt1_0_35 * (2 + (input[i] > (142+offset1)));
t1_0_35 += (1 - rt1_0_35) * (input[i] > (140+offset1));

int rt1_0_36 = (input[i] > (145+offset1));
int t1_0_36 = rt1_0_36 * (2 + (input[i] > (146+offset1)));
t1_0_36 += (1 - rt1_0_36) * (input[i] > (144+offset1));

int rt1_0_37 = (input[i] > (149+offset1));
int t1_0_37 = rt1_0_37 * (2 + (input[i] > (150+offset1)));
t1_0_37 += (1 - rt1_0_37) * (input[i] > (148+offset1));

int rt1_0_38 = (input[i] > (153+offset1));
int t1_0_38 = rt1_0_38 * (2 + (input[i] > (154+offset1)));
t1_0_38 += (1 - rt1_0_38) * (input[i] > (152+offset1));

int rt1_0_39 = (input[i] > (157+offset1));
int t1_0_39 = rt1_0_39 * (2 + (input[i] > (158+offset1)));
t1_0_39 += (1 - rt1_0_39) * (input[i] > (156+offset1));

int rt1_0_40 = (input[i] > (161+offset1));
int t1_0_40 = rt1_0_40 * (2 + (input[i] > (162+offset1)));
t1_0_40 += (1 - rt1_0_40) * (input[i] > (160+offset1));

int rt1_0_41 = (input[i] > (165+offset1));
int t1_0_41 = rt1_0_41 * (2 + (input[i] > (166+offset1)));
t1_0_41 += (1 - rt1_0_41) * (input[i] > (164+offset1));

int rt1_0_42 = (input[i] > (169+offset1));
int t1_0_42 = rt1_0_42 * (2 + (input[i] > (170+offset1)));
t1_0_42 += (1 - rt1_0_42) * (input[i] > (168+offset1));

int rt1_0_43 = (input[i] > (173+offset1));
int t1_0_43 = rt1_0_43 * (2 + (input[i] > (174+offset1)));
t1_0_43 += (1 - rt1_0_43) * (input[i] > (172+offset1));

int rt1_0_44 = (input[i] > (177+offset1));
int t1_0_44 = rt1_0_44 * (2 + (input[i] > (178+offset1)));
t1_0_44 += (1 - rt1_0_44) * (input[i] > (176+offset1));

int rt1_0_45 = (input[i] > (181+offset1));
int t1_0_45 = rt1_0_45 * (2 + (input[i] > (182+offset1)));
t1_0_45 += (1 - rt1_0_45) * (input[i] > (180+offset1));

int rt1_0_46 = (input[i] > (185+offset1));
int t1_0_46 = rt1_0_46 * (2 + (input[i] > (186+offset1)));
t1_0_46 += (1 - rt1_0_46) * (input[i] > (184+offset1));

int rt1_0_47 = (input[i] > (189+offset1));
int t1_0_47 = rt1_0_47 * (2 + (input[i] > (190+offset1)));
t1_0_47 += (1 - rt1_0_47) * (input[i] > (188+offset1));

int rt1_0_48 = (input[i] > (193+offset1));
int t1_0_48 = rt1_0_48 * (2 + (input[i] > (194+offset1)));
t1_0_48 += (1 - rt1_0_48) * (input[i] > (192+offset1));

int rt1_0_49 = (input[i] > (197+offset1));
int t1_0_49 = rt1_0_49 * (2 + (input[i] > (198+offset1)));
t1_0_49 += (1 - rt1_0_49) * (input[i] > (196+offset1));

int rt1_0_50 = (input[i] > (201+offset1));
int t1_0_50 = rt1_0_50 * (2 + (input[i] > (202+offset1)));
t1_0_50 += (1 - rt1_0_50) * (input[i] > (200+offset1));

int rt1_0_51 = (input[i] > (205+offset1));
int t1_0_51 = rt1_0_51 * (2 + (input[i] > (206+offset1)));
t1_0_51 += (1 - rt1_0_51) * (input[i] > (204+offset1));

int rt1_0_52 = (input[i] > (209+offset1));
int t1_0_52 = rt1_0_52 * (2 + (input[i] > (210+offset1)));
t1_0_52 += (1 - rt1_0_52) * (input[i] > (208+offset1));

int rt1_0_53 = (input[i] > (213+offset1));
int t1_0_53 = rt1_0_53 * (2 + (input[i] > (214+offset1)));
t1_0_53 += (1 - rt1_0_53) * (input[i] > (212+offset1));

int rt1_0_54 = (input[i] > (217+offset1));
int t1_0_54 = rt1_0_54 * (2 + (input[i] > (218+offset1)));
t1_0_54 += (1 - rt1_0_54) * (input[i] > (216+offset1));

int rt1_0_55 = (input[i] > (221+offset1));
int t1_0_55 = rt1_0_55 * (2 + (input[i] > (222+offset1)));
t1_0_55 += (1 - rt1_0_55) * (input[i] > (220+offset1));

int rt1_0_56 = (input[i] > (225+offset1));
int t1_0_56 = rt1_0_56 * (2 + (input[i] > (226+offset1)));
t1_0_56 += (1 - rt1_0_56) * (input[i] > (224+offset1));

int rt1_0_57 = (input[i] > (229+offset1));
int t1_0_57 = rt1_0_57 * (2 + (input[i] > (230+offset1)));
t1_0_57 += (1 - rt1_0_57) * (input[i] > (228+offset1));

int rt1_0_58 = (input[i] > (233+offset1));
int t1_0_58 = rt1_0_58 * (2 + (input[i] > (234+offset1)));
t1_0_58 += (1 - rt1_0_58) * (input[i] > (232+offset1));

int rt1_0_59 = (input[i] > (237+offset1));
int t1_0_59 = rt1_0_59 * (2 + (input[i] > (238+offset1)));
t1_0_59 += (1 - rt1_0_59) * (input[i] > (236+offset1));

int rt1_0_60 = (input[i] > (241+offset1));
int t1_0_60 = rt1_0_60 * (2 + (input[i] > (242+offset1)));
t1_0_60 += (1 - rt1_0_60) * (input[i] > (240+offset1));

int rt1_0_61 = (input[i] > (245+offset1));
int t1_0_61 = rt1_0_61 * (2 + (input[i] > (246+offset1)));
t1_0_61 += (1 - rt1_0_61) * (input[i] > (244+offset1));

int rt1_0_62 = (input[i] > (249+offset1));
int t1_0_62 = rt1_0_62 * (2 + (input[i] > (250+offset1)));
t1_0_62 += (1 - rt1_0_62) * (input[i] > (248+offset1));

int rt1_0_63 = (input[i] > (253+offset1));
int t1_0_63 = rt1_0_63 * (2 + (input[i] > (254+offset1)));
t1_0_63 += (1 - rt1_0_63) * (input[i] > (252+offset1));

int rt1_1_0 = (input[i] > (3+offset1));
int t1_1_0 = rt1_1_0 * (4 + t1_0_1);
t1_1_0 += (1 - rt1_1_0) * t1_0_0;

int rt1_1_1 = (input[i] > (11+offset1));
int t1_1_1 = rt1_1_1 * (4 + t1_0_3);
t1_1_1 += (1 - rt1_1_1) * t1_0_2;

int rt1_1_2 = (input[i] > (19+offset1));
int t1_1_2 = rt1_1_2 * (4 + t1_0_5);
t1_1_2 += (1 - rt1_1_2) * t1_0_4;

int rt1_1_3 = (input[i] > (27+offset1));
int t1_1_3 = rt1_1_3 * (4 + t1_0_7);
t1_1_3 += (1 - rt1_1_3) * t1_0_6;

int rt1_1_4 = (input[i] > (35+offset1));
int t1_1_4 = rt1_1_4 * (4 + t1_0_9);
t1_1_4 += (1 - rt1_1_4) * t1_0_8;

int rt1_1_5 = (input[i] > (43+offset1));
int t1_1_5 = rt1_1_5 * (4 + t1_0_11);
t1_1_5 += (1 - rt1_1_5) * t1_0_10;

int rt1_1_6 = (input[i] > (51+offset1));
int t1_1_6 = rt1_1_6 * (4 + t1_0_13);
t1_1_6 += (1 - rt1_1_6) * t1_0_12;

int rt1_1_7 = (input[i] > (59+offset1));
int t1_1_7 = rt1_1_7 * (4 + t1_0_15);
t1_1_7 += (1 - rt1_1_7) * t1_0_14;

int rt1_1_8 = (input[i] > (67+offset1));
int t1_1_8 = rt1_1_8 * (4 + t1_0_17);
t1_1_8 += (1 - rt1_1_8) * t1_0_16;

int rt1_1_9 = (input[i] > (75+offset1));
int t1_1_9 = rt1_1_9 * (4 + t1_0_19);
t1_1_9 += (1 - rt1_1_9) * t1_0_18;

int rt1_1_10 = (input[i] > (83+offset1));
int t1_1_10 = rt1_1_10 * (4 + t1_0_21);
t1_1_10 += (1 - rt1_1_10) * t1_0_20;

int rt1_1_11 = (input[i] > (91+offset1));
int t1_1_11 = rt1_1_11 * (4 + t1_0_23);
t1_1_11 += (1 - rt1_1_11) * t1_0_22;

int rt1_1_12 = (input[i] > (99+offset1));
int t1_1_12 = rt1_1_12 * (4 + t1_0_25);
t1_1_12 += (1 - rt1_1_12) * t1_0_24;

int rt1_1_13 = (input[i] > (107+offset1));
int t1_1_13 = rt1_1_13 * (4 + t1_0_27);
t1_1_13 += (1 - rt1_1_13) * t1_0_26;

int rt1_1_14 = (input[i] > (115+offset1));
int t1_1_14 = rt1_1_14 * (4 + t1_0_29);
t1_1_14 += (1 - rt1_1_14) * t1_0_28;

int rt1_1_15 = (input[i] > (123+offset1));
int t1_1_15 = rt1_1_15 * (4 + t1_0_31);
t1_1_15 += (1 - rt1_1_15) * t1_0_30;

int rt1_1_16 = (input[i] > (131+offset1));
int t1_1_16 = rt1_1_16 * (4 + t1_0_33);
t1_1_16 += (1 - rt1_1_16) * t1_0_32;

int rt1_1_17 = (input[i] > (139+offset1));
int t1_1_17 = rt1_1_17 * (4 + t1_0_35);
t1_1_17 += (1 - rt1_1_17) * t1_0_34;

int rt1_1_18 = (input[i] > (147+offset1));
int t1_1_18 = rt1_1_18 * (4 + t1_0_37);
t1_1_18 += (1 - rt1_1_18) * t1_0_36;

int rt1_1_19 = (input[i] > (155+offset1));
int t1_1_19 = rt1_1_19 * (4 + t1_0_39);
t1_1_19 += (1 - rt1_1_19) * t1_0_38;

int rt1_1_20 = (input[i] > (163+offset1));
int t1_1_20 = rt1_1_20 * (4 + t1_0_41);
t1_1_20 += (1 - rt1_1_20) * t1_0_40;

int rt1_1_21 = (input[i] > (171+offset1));
int t1_1_21 = rt1_1_21 * (4 + t1_0_43);
t1_1_21 += (1 - rt1_1_21) * t1_0_42;

int rt1_1_22 = (input[i] > (179+offset1));
int t1_1_22 = rt1_1_22 * (4 + t1_0_45);
t1_1_22 += (1 - rt1_1_22) * t1_0_44;

int rt1_1_23 = (input[i] > (187+offset1));
int t1_1_23 = rt1_1_23 * (4 + t1_0_47);
t1_1_23 += (1 - rt1_1_23) * t1_0_46;

int rt1_1_24 = (input[i] > (195+offset1));
int t1_1_24 = rt1_1_24 * (4 + t1_0_49);
t1_1_24 += (1 - rt1_1_24) * t1_0_48;

int rt1_1_25 = (input[i] > (203+offset1));
int t1_1_25 = rt1_1_25 * (4 + t1_0_51);
t1_1_25 += (1 - rt1_1_25) * t1_0_50;

int rt1_1_26 = (input[i] > (211+offset1));
int t1_1_26 = rt1_1_26 * (4 + t1_0_53);
t1_1_26 += (1 - rt1_1_26) * t1_0_52;

int rt1_1_27 = (input[i] > (219+offset1));
int t1_1_27 = rt1_1_27 * (4 + t1_0_55);
t1_1_27 += (1 - rt1_1_27) * t1_0_54;

int rt1_1_28 = (input[i] > (227+offset1));
int t1_1_28 = rt1_1_28 * (4 + t1_0_57);
t1_1_28 += (1 - rt1_1_28) * t1_0_56;

int rt1_1_29 = (input[i] > (235+offset1));
int t1_1_29 = rt1_1_29 * (4 + t1_0_59);
t1_1_29 += (1 - rt1_1_29) * t1_0_58;

int rt1_1_30 = (input[i] > (243+offset1));
int t1_1_30 = rt1_1_30 * (4 + t1_0_61);
t1_1_30 += (1 - rt1_1_30) * t1_0_60;

int rt1_1_31 = (input[i] > (251+offset1));
int t1_1_31 = rt1_1_31 * (4 + t1_0_63);
t1_1_31 += (1 - rt1_1_31) * t1_0_62;

int rt1_2_0 = (input[i] > (7+offset1));
int t1_2_0 = rt1_2_0 * (8 + t1_1_1);
t1_2_0 += (1 - rt1_2_0) * t1_1_0;

int rt1_2_1 = (input[i] > (23+offset1));
int t1_2_1 = rt1_2_1 * (8 + t1_1_3);
t1_2_1 += (1 - rt1_2_1) * t1_1_2;

int rt1_2_2 = (input[i] > (39+offset1));
int t1_2_2 = rt1_2_2 * (8 + t1_1_5);
t1_2_2 += (1 - rt1_2_2) * t1_1_4;

int rt1_2_3 = (input[i] > (55+offset1));
int t1_2_3 = rt1_2_3 * (8 + t1_1_7);
t1_2_3 += (1 - rt1_2_3) * t1_1_6;

int rt1_2_4 = (input[i] > (71+offset1));
int t1_2_4 = rt1_2_4 * (8 + t1_1_9);
t1_2_4 += (1 - rt1_2_4) * t1_1_8;

int rt1_2_5 = (input[i] > (87+offset1));
int t1_2_5 = rt1_2_5 * (8 + t1_1_11);
t1_2_5 += (1 - rt1_2_5) * t1_1_10;

int rt1_2_6 = (input[i] > (103+offset1));
int t1_2_6 = rt1_2_6 * (8 + t1_1_13);
t1_2_6 += (1 - rt1_2_6) * t1_1_12;

int rt1_2_7 = (input[i] > (119+offset1));
int t1_2_7 = rt1_2_7 * (8 + t1_1_15);
t1_2_7 += (1 - rt1_2_7) * t1_1_14;

int rt1_2_8 = (input[i] > (135+offset1));
int t1_2_8 = rt1_2_8 * (8 + t1_1_17);
t1_2_8 += (1 - rt1_2_8) * t1_1_16;

int rt1_2_9 = (input[i] > (151+offset1));
int t1_2_9 = rt1_2_9 * (8 + t1_1_19);
t1_2_9 += (1 - rt1_2_9) * t1_1_18;

int rt1_2_10 = (input[i] > (167+offset1));
int t1_2_10 = rt1_2_10 * (8 + t1_1_21);
t1_2_10 += (1 - rt1_2_10) * t1_1_20;

int rt1_2_11 = (input[i] > (183+offset1));
int t1_2_11 = rt1_2_11 * (8 + t1_1_23);
t1_2_11 += (1 - rt1_2_11) * t1_1_22;

int rt1_2_12 = (input[i] > (199+offset1));
int t1_2_12 = rt1_2_12 * (8 + t1_1_25);
t1_2_12 += (1 - rt1_2_12) * t1_1_24;

int rt1_2_13 = (input[i] > (215+offset1));
int t1_2_13 = rt1_2_13 * (8 + t1_1_27);
t1_2_13 += (1 - rt1_2_13) * t1_1_26;

int rt1_2_14 = (input[i] > (231+offset1));
int t1_2_14 = rt1_2_14 * (8 + t1_1_29);
t1_2_14 += (1 - rt1_2_14) * t1_1_28;

int rt1_2_15 = (input[i] > (247+offset1));
int t1_2_15 = rt1_2_15 * (8 + t1_1_31);
t1_2_15 += (1 - rt1_2_15) * t1_1_30;

int rt1_3_0 = (input[i] > (15+offset1));
int t1_3_0 = rt1_3_0 * (16 + t1_2_1);
t1_3_0 += (1 - rt1_3_0) * t1_2_0;

int rt1_3_1 = (input[i] > (47+offset1));
int t1_3_1 = rt1_3_1 * (16 + t1_2_3);
t1_3_1 += (1 - rt1_3_1) * t1_2_2;

int rt1_3_2 = (input[i] > (79+offset1));
int t1_3_2 = rt1_3_2 * (16 + t1_2_5);
t1_3_2 += (1 - rt1_3_2) * t1_2_4;

int rt1_3_3 = (input[i] > (111+offset1));
int t1_3_3 = rt1_3_3 * (16 + t1_2_7);
t1_3_3 += (1 - rt1_3_3) * t1_2_6;

int rt1_3_4 = (input[i] > (143+offset1));
int t1_3_4 = rt1_3_4 * (16 + t1_2_9);
t1_3_4 += (1 - rt1_3_4) * t1_2_8;

int rt1_3_5 = (input[i] > (175+offset1));
int t1_3_5 = rt1_3_5 * (16 + t1_2_11);
t1_3_5 += (1 - rt1_3_5) * t1_2_10;

int rt1_3_6 = (input[i] > (207+offset1));
int t1_3_6 = rt1_3_6 * (16 + t1_2_13);
t1_3_6 += (1 - rt1_3_6) * t1_2_12;

int rt1_3_7 = (input[i] > (239+offset1));
int t1_3_7 = rt1_3_7 * (16 + t1_2_15);
t1_3_7 += (1 - rt1_3_7) * t1_2_14;

int rt1_4_0 = (input[i] > (31+offset1));
int t1_4_0 = rt1_4_0 * (32 + t1_3_1);
t1_4_0 += (1 - rt1_4_0) * t1_3_0;

int rt1_4_1 = (input[i] > (95+offset1));
int t1_4_1 = rt1_4_1 * (32 + t1_3_3);
t1_4_1 += (1 - rt1_4_1) * t1_3_2;

int rt1_4_2 = (input[i] > (159+offset1));
int t1_4_2 = rt1_4_2 * (32 + t1_3_5);
t1_4_2 += (1 - rt1_4_2) * t1_3_4;

int rt1_4_3 = (input[i] > (223+offset1));
int t1_4_3 = rt1_4_3 * (32 + t1_3_7);
t1_4_3 += (1 - rt1_4_3) * t1_3_6;

int rt1_5_0 = (input[i] > (63+offset1));
int t1_5_0 = rt1_5_0 * (64 + t1_4_1);
t1_5_0 += (1 - rt1_5_0) * t1_4_0;

int rt1_5_1 = (input[i] > (191+offset1));
int t1_5_1 = rt1_5_1 * (64 + t1_4_3);
t1_5_1 += (1 - rt1_5_1) * t1_4_2;

int rt1_6_0 = (input[i] > (127+offset1));
int t1_6_0 = rt1_6_0 * (128 + t1_5_1);
t1_6_0 += (1 - rt1_6_0) * t1_5_0;

int rt2_0_0 = (input[i] > (1+offset2));
int t2_0_0 = rt2_0_0 * (2 + (input[i] > (2+offset2)));
t2_0_0 += (1 - rt2_0_0) * (input[i] > (0+offset2));

int rt2_0_1 = (input[i] > (5+offset2));
int t2_0_1 = rt2_0_1 * (2 + (input[i] > (6+offset2)));
t2_0_1 += (1 - rt2_0_1) * (input[i] > (4+offset2));

int rt2_0_2 = (input[i] > (9+offset2));
int t2_0_2 = rt2_0_2 * (2 + (input[i] > (10+offset2)));
t2_0_2 += (1 - rt2_0_2) * (input[i] > (8+offset2));

int rt2_0_3 = (input[i] > (13+offset2));
int t2_0_3 = rt2_0_3 * (2 + (input[i] > (14+offset2)));
t2_0_3 += (1 - rt2_0_3) * (input[i] > (12+offset2));

int rt2_0_4 = (input[i] > (17+offset2));
int t2_0_4 = rt2_0_4 * (2 + (input[i] > (18+offset2)));
t2_0_4 += (1 - rt2_0_4) * (input[i] > (16+offset2));

int rt2_0_5 = (input[i] > (21+offset2));
int t2_0_5 = rt2_0_5 * (2 + (input[i] > (22+offset2)));
t2_0_5 += (1 - rt2_0_5) * (input[i] > (20+offset2));

int rt2_0_6 = (input[i] > (25+offset2));
int t2_0_6 = rt2_0_6 * (2 + (input[i] > (26+offset2)));
t2_0_6 += (1 - rt2_0_6) * (input[i] > (24+offset2));

int rt2_0_7 = (input[i] > (29+offset2));
int t2_0_7 = rt2_0_7 * (2 + (input[i] > (30+offset2)));
t2_0_7 += (1 - rt2_0_7) * (input[i] > (28+offset2));

int rt2_0_8 = (input[i] > (33+offset2));
int t2_0_8 = rt2_0_8 * (2 + (input[i] > (34+offset2)));
t2_0_8 += (1 - rt2_0_8) * (input[i] > (32+offset2));

int rt2_0_9 = (input[i] > (37+offset2));
int t2_0_9 = rt2_0_9 * (2 + (input[i] > (38+offset2)));
t2_0_9 += (1 - rt2_0_9) * (input[i] > (36+offset2));

int rt2_0_10 = (input[i] > (41+offset2));
int t2_0_10 = rt2_0_10 * (2 + (input[i] > (42+offset2)));
t2_0_10 += (1 - rt2_0_10) * (input[i] > (40+offset2));

int rt2_0_11 = (input[i] > (45+offset2));
int t2_0_11 = rt2_0_11 * (2 + (input[i] > (46+offset2)));
t2_0_11 += (1 - rt2_0_11) * (input[i] > (44+offset2));

int rt2_0_12 = (input[i] > (49+offset2));
int t2_0_12 = rt2_0_12 * (2 + (input[i] > (50+offset2)));
t2_0_12 += (1 - rt2_0_12) * (input[i] > (48+offset2));

int rt2_0_13 = (input[i] > (53+offset2));
int t2_0_13 = rt2_0_13 * (2 + (input[i] > (54+offset2)));
t2_0_13 += (1 - rt2_0_13) * (input[i] > (52+offset2));

int rt2_0_14 = (input[i] > (57+offset2));
int t2_0_14 = rt2_0_14 * (2 + (input[i] > (58+offset2)));
t2_0_14 += (1 - rt2_0_14) * (input[i] > (56+offset2));

int rt2_0_15 = (input[i] > (61+offset2));
int t2_0_15 = rt2_0_15 * (2 + (input[i] > (62+offset2)));
t2_0_15 += (1 - rt2_0_15) * (input[i] > (60+offset2));

int rt2_0_16 = (input[i] > (65+offset2));
int t2_0_16 = rt2_0_16 * (2 + (input[i] > (66+offset2)));
t2_0_16 += (1 - rt2_0_16) * (input[i] > (64+offset2));

int rt2_0_17 = (input[i] > (69+offset2));
int t2_0_17 = rt2_0_17 * (2 + (input[i] > (70+offset2)));
t2_0_17 += (1 - rt2_0_17) * (input[i] > (68+offset2));

int rt2_0_18 = (input[i] > (73+offset2));
int t2_0_18 = rt2_0_18 * (2 + (input[i] > (74+offset2)));
t2_0_18 += (1 - rt2_0_18) * (input[i] > (72+offset2));

int rt2_0_19 = (input[i] > (77+offset2));
int t2_0_19 = rt2_0_19 * (2 + (input[i] > (78+offset2)));
t2_0_19 += (1 - rt2_0_19) * (input[i] > (76+offset2));

int rt2_0_20 = (input[i] > (81+offset2));
int t2_0_20 = rt2_0_20 * (2 + (input[i] > (82+offset2)));
t2_0_20 += (1 - rt2_0_20) * (input[i] > (80+offset2));

int rt2_0_21 = (input[i] > (85+offset2));
int t2_0_21 = rt2_0_21 * (2 + (input[i] > (86+offset2)));
t2_0_21 += (1 - rt2_0_21) * (input[i] > (84+offset2));

int rt2_0_22 = (input[i] > (89+offset2));
int t2_0_22 = rt2_0_22 * (2 + (input[i] > (90+offset2)));
t2_0_22 += (1 - rt2_0_22) * (input[i] > (88+offset2));

int rt2_0_23 = (input[i] > (93+offset2));
int t2_0_23 = rt2_0_23 * (2 + (input[i] > (94+offset2)));
t2_0_23 += (1 - rt2_0_23) * (input[i] > (92+offset2));

int rt2_0_24 = (input[i] > (97+offset2));
int t2_0_24 = rt2_0_24 * (2 + (input[i] > (98+offset2)));
t2_0_24 += (1 - rt2_0_24) * (input[i] > (96+offset2));

int rt2_0_25 = (input[i] > (101+offset2));
int t2_0_25 = rt2_0_25 * (2 + (input[i] > (102+offset2)));
t2_0_25 += (1 - rt2_0_25) * (input[i] > (100+offset2));

int rt2_0_26 = (input[i] > (105+offset2));
int t2_0_26 = rt2_0_26 * (2 + (input[i] > (106+offset2)));
t2_0_26 += (1 - rt2_0_26) * (input[i] > (104+offset2));

int rt2_0_27 = (input[i] > (109+offset2));
int t2_0_27 = rt2_0_27 * (2 + (input[i] > (110+offset2)));
t2_0_27 += (1 - rt2_0_27) * (input[i] > (108+offset2));

int rt2_0_28 = (input[i] > (113+offset2));
int t2_0_28 = rt2_0_28 * (2 + (input[i] > (114+offset2)));
t2_0_28 += (1 - rt2_0_28) * (input[i] > (112+offset2));

int rt2_0_29 = (input[i] > (117+offset2));
int t2_0_29 = rt2_0_29 * (2 + (input[i] > (118+offset2)));
t2_0_29 += (1 - rt2_0_29) * (input[i] > (116+offset2));

int rt2_0_30 = (input[i] > (121+offset2));
int t2_0_30 = rt2_0_30 * (2 + (input[i] > (122+offset2)));
t2_0_30 += (1 - rt2_0_30) * (input[i] > (120+offset2));

int rt2_0_31 = (input[i] > (125+offset2));
int t2_0_31 = rt2_0_31 * (2 + (input[i] > (126+offset2)));
t2_0_31 += (1 - rt2_0_31) * (input[i] > (124+offset2));

int rt2_0_32 = (input[i] > (129+offset2));
int t2_0_32 = rt2_0_32 * (2 + (input[i] > (130+offset2)));
t2_0_32 += (1 - rt2_0_32) * (input[i] > (128+offset2));

int rt2_0_33 = (input[i] > (133+offset2));
int t2_0_33 = rt2_0_33 * (2 + (input[i] > (134+offset2)));
t2_0_33 += (1 - rt2_0_33) * (input[i] > (132+offset2));

int rt2_0_34 = (input[i] > (137+offset2));
int t2_0_34 = rt2_0_34 * (2 + (input[i] > (138+offset2)));
t2_0_34 += (1 - rt2_0_34) * (input[i] > (136+offset2));

int rt2_0_35 = (input[i] > (141+offset2));
int t2_0_35 = rt2_0_35 * (2 + (input[i] > (142+offset2)));
t2_0_35 += (1 - rt2_0_35) * (input[i] > (140+offset2));

int rt2_0_36 = (input[i] > (145+offset2));
int t2_0_36 = rt2_0_36 * (2 + (input[i] > (146+offset2)));
t2_0_36 += (1 - rt2_0_36) * (input[i] > (144+offset2));

int rt2_0_37 = (input[i] > (149+offset2));
int t2_0_37 = rt2_0_37 * (2 + (input[i] > (150+offset2)));
t2_0_37 += (1 - rt2_0_37) * (input[i] > (148+offset2));

int rt2_0_38 = (input[i] > (153+offset2));
int t2_0_38 = rt2_0_38 * (2 + (input[i] > (154+offset2)));
t2_0_38 += (1 - rt2_0_38) * (input[i] > (152+offset2));

int rt2_0_39 = (input[i] > (157+offset2));
int t2_0_39 = rt2_0_39 * (2 + (input[i] > (158+offset2)));
t2_0_39 += (1 - rt2_0_39) * (input[i] > (156+offset2));

int rt2_0_40 = (input[i] > (161+offset2));
int t2_0_40 = rt2_0_40 * (2 + (input[i] > (162+offset2)));
t2_0_40 += (1 - rt2_0_40) * (input[i] > (160+offset2));

int rt2_0_41 = (input[i] > (165+offset2));
int t2_0_41 = rt2_0_41 * (2 + (input[i] > (166+offset2)));
t2_0_41 += (1 - rt2_0_41) * (input[i] > (164+offset2));

int rt2_0_42 = (input[i] > (169+offset2));
int t2_0_42 = rt2_0_42 * (2 + (input[i] > (170+offset2)));
t2_0_42 += (1 - rt2_0_42) * (input[i] > (168+offset2));

int rt2_0_43 = (input[i] > (173+offset2));
int t2_0_43 = rt2_0_43 * (2 + (input[i] > (174+offset2)));
t2_0_43 += (1 - rt2_0_43) * (input[i] > (172+offset2));

int rt2_0_44 = (input[i] > (177+offset2));
int t2_0_44 = rt2_0_44 * (2 + (input[i] > (178+offset2)));
t2_0_44 += (1 - rt2_0_44) * (input[i] > (176+offset2));

int rt2_0_45 = (input[i] > (181+offset2));
int t2_0_45 = rt2_0_45 * (2 + (input[i] > (182+offset2)));
t2_0_45 += (1 - rt2_0_45) * (input[i] > (180+offset2));

int rt2_0_46 = (input[i] > (185+offset2));
int t2_0_46 = rt2_0_46 * (2 + (input[i] > (186+offset2)));
t2_0_46 += (1 - rt2_0_46) * (input[i] > (184+offset2));

int rt2_0_47 = (input[i] > (189+offset2));
int t2_0_47 = rt2_0_47 * (2 + (input[i] > (190+offset2)));
t2_0_47 += (1 - rt2_0_47) * (input[i] > (188+offset2));

int rt2_0_48 = (input[i] > (193+offset2));
int t2_0_48 = rt2_0_48 * (2 + (input[i] > (194+offset2)));
t2_0_48 += (1 - rt2_0_48) * (input[i] > (192+offset2));

int rt2_0_49 = (input[i] > (197+offset2));
int t2_0_49 = rt2_0_49 * (2 + (input[i] > (198+offset2)));
t2_0_49 += (1 - rt2_0_49) * (input[i] > (196+offset2));

int rt2_0_50 = (input[i] > (201+offset2));
int t2_0_50 = rt2_0_50 * (2 + (input[i] > (202+offset2)));
t2_0_50 += (1 - rt2_0_50) * (input[i] > (200+offset2));

int rt2_0_51 = (input[i] > (205+offset2));
int t2_0_51 = rt2_0_51 * (2 + (input[i] > (206+offset2)));
t2_0_51 += (1 - rt2_0_51) * (input[i] > (204+offset2));

int rt2_0_52 = (input[i] > (209+offset2));
int t2_0_52 = rt2_0_52 * (2 + (input[i] > (210+offset2)));
t2_0_52 += (1 - rt2_0_52) * (input[i] > (208+offset2));

int rt2_0_53 = (input[i] > (213+offset2));
int t2_0_53 = rt2_0_53 * (2 + (input[i] > (214+offset2)));
t2_0_53 += (1 - rt2_0_53) * (input[i] > (212+offset2));

int rt2_0_54 = (input[i] > (217+offset2));
int t2_0_54 = rt2_0_54 * (2 + (input[i] > (218+offset2)));
t2_0_54 += (1 - rt2_0_54) * (input[i] > (216+offset2));

int rt2_0_55 = (input[i] > (221+offset2));
int t2_0_55 = rt2_0_55 * (2 + (input[i] > (222+offset2)));
t2_0_55 += (1 - rt2_0_55) * (input[i] > (220+offset2));

int rt2_0_56 = (input[i] > (225+offset2));
int t2_0_56 = rt2_0_56 * (2 + (input[i] > (226+offset2)));
t2_0_56 += (1 - rt2_0_56) * (input[i] > (224+offset2));

int rt2_0_57 = (input[i] > (229+offset2));
int t2_0_57 = rt2_0_57 * (2 + (input[i] > (230+offset2)));
t2_0_57 += (1 - rt2_0_57) * (input[i] > (228+offset2));

int rt2_0_58 = (input[i] > (233+offset2));
int t2_0_58 = rt2_0_58 * (2 + (input[i] > (234+offset2)));
t2_0_58 += (1 - rt2_0_58) * (input[i] > (232+offset2));

int rt2_0_59 = (input[i] > (237+offset2));
int t2_0_59 = rt2_0_59 * (2 + (input[i] > (238+offset2)));
t2_0_59 += (1 - rt2_0_59) * (input[i] > (236+offset2));

int rt2_0_60 = (input[i] > (241+offset2));
int t2_0_60 = rt2_0_60 * (2 + (input[i] > (242+offset2)));
t2_0_60 += (1 - rt2_0_60) * (input[i] > (240+offset2));

int rt2_0_61 = (input[i] > (245+offset2));
int t2_0_61 = rt2_0_61 * (2 + (input[i] > (246+offset2)));
t2_0_61 += (1 - rt2_0_61) * (input[i] > (244+offset2));

int rt2_0_62 = (input[i] > (249+offset2));
int t2_0_62 = rt2_0_62 * (2 + (input[i] > (250+offset2)));
t2_0_62 += (1 - rt2_0_62) * (input[i] > (248+offset2));

int rt2_0_63 = (input[i] > (253+offset2));
int t2_0_63 = rt2_0_63 * (2 + (input[i] > (254+offset2)));
t2_0_63 += (1 - rt2_0_63) * (input[i] > (252+offset2));

int rt2_1_0 = (input[i] > (3+offset2));
int t2_1_0 = rt2_1_0 * (4 + t2_0_1);
t2_1_0 += (1 - rt2_1_0) * t2_0_0;

int rt2_1_1 = (input[i] > (11+offset2));
int t2_1_1 = rt2_1_1 * (4 + t2_0_3);
t2_1_1 += (1 - rt2_1_1) * t2_0_2;

int rt2_1_2 = (input[i] > (19+offset2));
int t2_1_2 = rt2_1_2 * (4 + t2_0_5);
t2_1_2 += (1 - rt2_1_2) * t2_0_4;

int rt2_1_3 = (input[i] > (27+offset2));
int t2_1_3 = rt2_1_3 * (4 + t2_0_7);
t2_1_3 += (1 - rt2_1_3) * t2_0_6;

int rt2_1_4 = (input[i] > (35+offset2));
int t2_1_4 = rt2_1_4 * (4 + t2_0_9);
t2_1_4 += (1 - rt2_1_4) * t2_0_8;

int rt2_1_5 = (input[i] > (43+offset2));
int t2_1_5 = rt2_1_5 * (4 + t2_0_11);
t2_1_5 += (1 - rt2_1_5) * t2_0_10;

int rt2_1_6 = (input[i] > (51+offset2));
int t2_1_6 = rt2_1_6 * (4 + t2_0_13);
t2_1_6 += (1 - rt2_1_6) * t2_0_12;

int rt2_1_7 = (input[i] > (59+offset2));
int t2_1_7 = rt2_1_7 * (4 + t2_0_15);
t2_1_7 += (1 - rt2_1_7) * t2_0_14;

int rt2_1_8 = (input[i] > (67+offset2));
int t2_1_8 = rt2_1_8 * (4 + t2_0_17);
t2_1_8 += (1 - rt2_1_8) * t2_0_16;

int rt2_1_9 = (input[i] > (75+offset2));
int t2_1_9 = rt2_1_9 * (4 + t2_0_19);
t2_1_9 += (1 - rt2_1_9) * t2_0_18;

int rt2_1_10 = (input[i] > (83+offset2));
int t2_1_10 = rt2_1_10 * (4 + t2_0_21);
t2_1_10 += (1 - rt2_1_10) * t2_0_20;

int rt2_1_11 = (input[i] > (91+offset2));
int t2_1_11 = rt2_1_11 * (4 + t2_0_23);
t2_1_11 += (1 - rt2_1_11) * t2_0_22;

int rt2_1_12 = (input[i] > (99+offset2));
int t2_1_12 = rt2_1_12 * (4 + t2_0_25);
t2_1_12 += (1 - rt2_1_12) * t2_0_24;

int rt2_1_13 = (input[i] > (107+offset2));
int t2_1_13 = rt2_1_13 * (4 + t2_0_27);
t2_1_13 += (1 - rt2_1_13) * t2_0_26;

int rt2_1_14 = (input[i] > (115+offset2));
int t2_1_14 = rt2_1_14 * (4 + t2_0_29);
t2_1_14 += (1 - rt2_1_14) * t2_0_28;

int rt2_1_15 = (input[i] > (123+offset2));
int t2_1_15 = rt2_1_15 * (4 + t2_0_31);
t2_1_15 += (1 - rt2_1_15) * t2_0_30;

int rt2_1_16 = (input[i] > (131+offset2));
int t2_1_16 = rt2_1_16 * (4 + t2_0_33);
t2_1_16 += (1 - rt2_1_16) * t2_0_32;

int rt2_1_17 = (input[i] > (139+offset2));
int t2_1_17 = rt2_1_17 * (4 + t2_0_35);
t2_1_17 += (1 - rt2_1_17) * t2_0_34;

int rt2_1_18 = (input[i] > (147+offset2));
int t2_1_18 = rt2_1_18 * (4 + t2_0_37);
t2_1_18 += (1 - rt2_1_18) * t2_0_36;

int rt2_1_19 = (input[i] > (155+offset2));
int t2_1_19 = rt2_1_19 * (4 + t2_0_39);
t2_1_19 += (1 - rt2_1_19) * t2_0_38;

int rt2_1_20 = (input[i] > (163+offset2));
int t2_1_20 = rt2_1_20 * (4 + t2_0_41);
t2_1_20 += (1 - rt2_1_20) * t2_0_40;

int rt2_1_21 = (input[i] > (171+offset2));
int t2_1_21 = rt2_1_21 * (4 + t2_0_43);
t2_1_21 += (1 - rt2_1_21) * t2_0_42;

int rt2_1_22 = (input[i] > (179+offset2));
int t2_1_22 = rt2_1_22 * (4 + t2_0_45);
t2_1_22 += (1 - rt2_1_22) * t2_0_44;

int rt2_1_23 = (input[i] > (187+offset2));
int t2_1_23 = rt2_1_23 * (4 + t2_0_47);
t2_1_23 += (1 - rt2_1_23) * t2_0_46;

int rt2_1_24 = (input[i] > (195+offset2));
int t2_1_24 = rt2_1_24 * (4 + t2_0_49);
t2_1_24 += (1 - rt2_1_24) * t2_0_48;

int rt2_1_25 = (input[i] > (203+offset2));
int t2_1_25 = rt2_1_25 * (4 + t2_0_51);
t2_1_25 += (1 - rt2_1_25) * t2_0_50;

int rt2_1_26 = (input[i] > (211+offset2));
int t2_1_26 = rt2_1_26 * (4 + t2_0_53);
t2_1_26 += (1 - rt2_1_26) * t2_0_52;

int rt2_1_27 = (input[i] > (219+offset2));
int t2_1_27 = rt2_1_27 * (4 + t2_0_55);
t2_1_27 += (1 - rt2_1_27) * t2_0_54;

int rt2_1_28 = (input[i] > (227+offset2));
int t2_1_28 = rt2_1_28 * (4 + t2_0_57);
t2_1_28 += (1 - rt2_1_28) * t2_0_56;

int rt2_1_29 = (input[i] > (235+offset2));
int t2_1_29 = rt2_1_29 * (4 + t2_0_59);
t2_1_29 += (1 - rt2_1_29) * t2_0_58;

int rt2_1_30 = (input[i] > (243+offset2));
int t2_1_30 = rt2_1_30 * (4 + t2_0_61);
t2_1_30 += (1 - rt2_1_30) * t2_0_60;

int rt2_1_31 = (input[i] > (251+offset2));
int t2_1_31 = rt2_1_31 * (4 + t2_0_63);
t2_1_31 += (1 - rt2_1_31) * t2_0_62;

int rt2_2_0 = (input[i] > (7+offset2));
int t2_2_0 = rt2_2_0 * (8 + t2_1_1);
t2_2_0 += (1 - rt2_2_0) * t2_1_0;

int rt2_2_1 = (input[i] > (23+offset2));
int t2_2_1 = rt2_2_1 * (8 + t2_1_3);
t2_2_1 += (1 - rt2_2_1) * t2_1_2;

int rt2_2_2 = (input[i] > (39+offset2));
int t2_2_2 = rt2_2_2 * (8 + t2_1_5);
t2_2_2 += (1 - rt2_2_2) * t2_1_4;

int rt2_2_3 = (input[i] > (55+offset2));
int t2_2_3 = rt2_2_3 * (8 + t2_1_7);
t2_2_3 += (1 - rt2_2_3) * t2_1_6;

int rt2_2_4 = (input[i] > (71+offset2));
int t2_2_4 = rt2_2_4 * (8 + t2_1_9);
t2_2_4 += (1 - rt2_2_4) * t2_1_8;

int rt2_2_5 = (input[i] > (87+offset2));
int t2_2_5 = rt2_2_5 * (8 + t2_1_11);
t2_2_5 += (1 - rt2_2_5) * t2_1_10;

int rt2_2_6 = (input[i] > (103+offset2));
int t2_2_6 = rt2_2_6 * (8 + t2_1_13);
t2_2_6 += (1 - rt2_2_6) * t2_1_12;

int rt2_2_7 = (input[i] > (119+offset2));
int t2_2_7 = rt2_2_7 * (8 + t2_1_15);
t2_2_7 += (1 - rt2_2_7) * t2_1_14;

int rt2_2_8 = (input[i] > (135+offset2));
int t2_2_8 = rt2_2_8 * (8 + t2_1_17);
t2_2_8 += (1 - rt2_2_8) * t2_1_16;

int rt2_2_9 = (input[i] > (151+offset2));
int t2_2_9 = rt2_2_9 * (8 + t2_1_19);
t2_2_9 += (1 - rt2_2_9) * t2_1_18;

int rt2_2_10 = (input[i] > (167+offset2));
int t2_2_10 = rt2_2_10 * (8 + t2_1_21);
t2_2_10 += (1 - rt2_2_10) * t2_1_20;

int rt2_2_11 = (input[i] > (183+offset2));
int t2_2_11 = rt2_2_11 * (8 + t2_1_23);
t2_2_11 += (1 - rt2_2_11) * t2_1_22;

int rt2_2_12 = (input[i] > (199+offset2));
int t2_2_12 = rt2_2_12 * (8 + t2_1_25);
t2_2_12 += (1 - rt2_2_12) * t2_1_24;

int rt2_2_13 = (input[i] > (215+offset2));
int t2_2_13 = rt2_2_13 * (8 + t2_1_27);
t2_2_13 += (1 - rt2_2_13) * t2_1_26;

int rt2_2_14 = (input[i] > (231+offset2));
int t2_2_14 = rt2_2_14 * (8 + t2_1_29);
t2_2_14 += (1 - rt2_2_14) * t2_1_28;

int rt2_2_15 = (input[i] > (247+offset2));
int t2_2_15 = rt2_2_15 * (8 + t2_1_31);
t2_2_15 += (1 - rt2_2_15) * t2_1_30;

int rt2_3_0 = (input[i] > (15+offset2));
int t2_3_0 = rt2_3_0 * (16 + t2_2_1);
t2_3_0 += (1 - rt2_3_0) * t2_2_0;

int rt2_3_1 = (input[i] > (47+offset2));
int t2_3_1 = rt2_3_1 * (16 + t2_2_3);
t2_3_1 += (1 - rt2_3_1) * t2_2_2;

int rt2_3_2 = (input[i] > (79+offset2));
int t2_3_2 = rt2_3_2 * (16 + t2_2_5);
t2_3_2 += (1 - rt2_3_2) * t2_2_4;

int rt2_3_3 = (input[i] > (111+offset2));
int t2_3_3 = rt2_3_3 * (16 + t2_2_7);
t2_3_3 += (1 - rt2_3_3) * t2_2_6;

int rt2_3_4 = (input[i] > (143+offset2));
int t2_3_4 = rt2_3_4 * (16 + t2_2_9);
t2_3_4 += (1 - rt2_3_4) * t2_2_8;

int rt2_3_5 = (input[i] > (175+offset2));
int t2_3_5 = rt2_3_5 * (16 + t2_2_11);
t2_3_5 += (1 - rt2_3_5) * t2_2_10;

int rt2_3_6 = (input[i] > (207+offset2));
int t2_3_6 = rt2_3_6 * (16 + t2_2_13);
t2_3_6 += (1 - rt2_3_6) * t2_2_12;

int rt2_3_7 = (input[i] > (239+offset2));
int t2_3_7 = rt2_3_7 * (16 + t2_2_15);
t2_3_7 += (1 - rt2_3_7) * t2_2_14;

int rt2_4_0 = (input[i] > (31+offset2));
int t2_4_0 = rt2_4_0 * (32 + t2_3_1);
t2_4_0 += (1 - rt2_4_0) * t2_3_0;

int rt2_4_1 = (input[i] > (95+offset2));
int t2_4_1 = rt2_4_1 * (32 + t2_3_3);
t2_4_1 += (1 - rt2_4_1) * t2_3_2;

int rt2_4_2 = (input[i] > (159+offset2));
int t2_4_2 = rt2_4_2 * (32 + t2_3_5);
t2_4_2 += (1 - rt2_4_2) * t2_3_4;

int rt2_4_3 = (input[i] > (223+offset2));
int t2_4_3 = rt2_4_3 * (32 + t2_3_7);
t2_4_3 += (1 - rt2_4_3) * t2_3_6;

int rt2_5_0 = (input[i] > (63+offset2));
int t2_5_0 = rt2_5_0 * (64 + t2_4_1);
t2_5_0 += (1 - rt2_5_0) * t2_4_0;

int rt2_5_1 = (input[i] > (191+offset2));
int t2_5_1 = rt2_5_1 * (64 + t2_4_3);
t2_5_1 += (1 - rt2_5_1) * t2_4_2;

int rt2_6_0 = (input[i] > (127+offset2));
int t2_6_0 = rt2_6_0 * (128 + t2_5_1);
t2_6_0 += (1 - rt2_6_0) * t2_5_0;

int rt3_0_0 = (input[i] > (1+offset3));
int t3_0_0 = rt3_0_0 * (2 + (input[i] > (2+offset3)));
t3_0_0 += (1 - rt3_0_0) * (input[i] > (0+offset3));

int rt3_0_1 = (input[i] > (5+offset3));
int t3_0_1 = rt3_0_1 * (2 + (input[i] > (6+offset3)));
t3_0_1 += (1 - rt3_0_1) * (input[i] > (4+offset3));

int rt3_0_2 = (input[i] > (9+offset3));
int t3_0_2 = rt3_0_2 * (2 + (input[i] > (10+offset3)));
t3_0_2 += (1 - rt3_0_2) * (input[i] > (8+offset3));

int rt3_0_3 = (input[i] > (13+offset3));
int t3_0_3 = rt3_0_3 * (2 + (input[i] > (14+offset3)));
t3_0_3 += (1 - rt3_0_3) * (input[i] > (12+offset3));

int rt3_0_4 = (input[i] > (17+offset3));
int t3_0_4 = rt3_0_4 * (2 + (input[i] > (18+offset3)));
t3_0_4 += (1 - rt3_0_4) * (input[i] > (16+offset3));

int rt3_0_5 = (input[i] > (21+offset3));
int t3_0_5 = rt3_0_5 * (2 + (input[i] > (22+offset3)));
t3_0_5 += (1 - rt3_0_5) * (input[i] > (20+offset3));

int rt3_0_6 = (input[i] > (25+offset3));
int t3_0_6 = rt3_0_6 * (2 + (input[i] > (26+offset3)));
t3_0_6 += (1 - rt3_0_6) * (input[i] > (24+offset3));

int rt3_0_7 = (input[i] > (29+offset3));
int t3_0_7 = rt3_0_7 * (2 + (input[i] > (30+offset3)));
t3_0_7 += (1 - rt3_0_7) * (input[i] > (28+offset3));

int rt3_0_8 = (input[i] > (33+offset3));
int t3_0_8 = rt3_0_8 * (2 + (input[i] > (34+offset3)));
t3_0_8 += (1 - rt3_0_8) * (input[i] > (32+offset3));

int rt3_0_9 = (input[i] > (37+offset3));
int t3_0_9 = rt3_0_9 * (2 + (input[i] > (38+offset3)));
t3_0_9 += (1 - rt3_0_9) * (input[i] > (36+offset3));

int rt3_0_10 = (input[i] > (41+offset3));
int t3_0_10 = rt3_0_10 * (2 + (input[i] > (42+offset3)));
t3_0_10 += (1 - rt3_0_10) * (input[i] > (40+offset3));

int rt3_0_11 = (input[i] > (45+offset3));
int t3_0_11 = rt3_0_11 * (2 + (input[i] > (46+offset3)));
t3_0_11 += (1 - rt3_0_11) * (input[i] > (44+offset3));

int rt3_0_12 = (input[i] > (49+offset3));
int t3_0_12 = rt3_0_12 * (2 + (input[i] > (50+offset3)));
t3_0_12 += (1 - rt3_0_12) * (input[i] > (48+offset3));

int rt3_0_13 = (input[i] > (53+offset3));
int t3_0_13 = rt3_0_13 * (2 + (input[i] > (54+offset3)));
t3_0_13 += (1 - rt3_0_13) * (input[i] > (52+offset3));

int rt3_0_14 = (input[i] > (57+offset3));
int t3_0_14 = rt3_0_14 * (2 + (input[i] > (58+offset3)));
t3_0_14 += (1 - rt3_0_14) * (input[i] > (56+offset3));

int rt3_0_15 = (input[i] > (61+offset3));
int t3_0_15 = rt3_0_15 * (2 + (input[i] > (62+offset3)));
t3_0_15 += (1 - rt3_0_15) * (input[i] > (60+offset3));

int rt3_0_16 = (input[i] > (65+offset3));
int t3_0_16 = rt3_0_16 * (2 + (input[i] > (66+offset3)));
t3_0_16 += (1 - rt3_0_16) * (input[i] > (64+offset3));

int rt3_0_17 = (input[i] > (69+offset3));
int t3_0_17 = rt3_0_17 * (2 + (input[i] > (70+offset3)));
t3_0_17 += (1 - rt3_0_17) * (input[i] > (68+offset3));

int rt3_0_18 = (input[i] > (73+offset3));
int t3_0_18 = rt3_0_18 * (2 + (input[i] > (74+offset3)));
t3_0_18 += (1 - rt3_0_18) * (input[i] > (72+offset3));

int rt3_0_19 = (input[i] > (77+offset3));
int t3_0_19 = rt3_0_19 * (2 + (input[i] > (78+offset3)));
t3_0_19 += (1 - rt3_0_19) * (input[i] > (76+offset3));

int rt3_0_20 = (input[i] > (81+offset3));
int t3_0_20 = rt3_0_20 * (2 + (input[i] > (82+offset3)));
t3_0_20 += (1 - rt3_0_20) * (input[i] > (80+offset3));

int rt3_0_21 = (input[i] > (85+offset3));
int t3_0_21 = rt3_0_21 * (2 + (input[i] > (86+offset3)));
t3_0_21 += (1 - rt3_0_21) * (input[i] > (84+offset3));

int rt3_0_22 = (input[i] > (89+offset3));
int t3_0_22 = rt3_0_22 * (2 + (input[i] > (90+offset3)));
t3_0_22 += (1 - rt3_0_22) * (input[i] > (88+offset3));

int rt3_0_23 = (input[i] > (93+offset3));
int t3_0_23 = rt3_0_23 * (2 + (input[i] > (94+offset3)));
t3_0_23 += (1 - rt3_0_23) * (input[i] > (92+offset3));

int rt3_0_24 = (input[i] > (97+offset3));
int t3_0_24 = rt3_0_24 * (2 + (input[i] > (98+offset3)));
t3_0_24 += (1 - rt3_0_24) * (input[i] > (96+offset3));

int rt3_0_25 = (input[i] > (101+offset3));
int t3_0_25 = rt3_0_25 * (2 + (input[i] > (102+offset3)));
t3_0_25 += (1 - rt3_0_25) * (input[i] > (100+offset3));

int rt3_0_26 = (input[i] > (105+offset3));
int t3_0_26 = rt3_0_26 * (2 + (input[i] > (106+offset3)));
t3_0_26 += (1 - rt3_0_26) * (input[i] > (104+offset3));

int rt3_0_27 = (input[i] > (109+offset3));
int t3_0_27 = rt3_0_27 * (2 + (input[i] > (110+offset3)));
t3_0_27 += (1 - rt3_0_27) * (input[i] > (108+offset3));

int rt3_0_28 = (input[i] > (113+offset3));
int t3_0_28 = rt3_0_28 * (2 + (input[i] > (114+offset3)));
t3_0_28 += (1 - rt3_0_28) * (input[i] > (112+offset3));

int rt3_0_29 = (input[i] > (117+offset3));
int t3_0_29 = rt3_0_29 * (2 + (input[i] > (118+offset3)));
t3_0_29 += (1 - rt3_0_29) * (input[i] > (116+offset3));

int rt3_0_30 = (input[i] > (121+offset3));
int t3_0_30 = rt3_0_30 * (2 + (input[i] > (122+offset3)));
t3_0_30 += (1 - rt3_0_30) * (input[i] > (120+offset3));

int rt3_0_31 = (input[i] > (125+offset3));
int t3_0_31 = rt3_0_31 * (2 + (input[i] > (126+offset3)));
t3_0_31 += (1 - rt3_0_31) * (input[i] > (124+offset3));

int rt3_0_32 = (input[i] > (129+offset3));
int t3_0_32 = rt3_0_32 * (2 + (input[i] > (130+offset3)));
t3_0_32 += (1 - rt3_0_32) * (input[i] > (128+offset3));

int rt3_0_33 = (input[i] > (133+offset3));
int t3_0_33 = rt3_0_33 * (2 + (input[i] > (134+offset3)));
t3_0_33 += (1 - rt3_0_33) * (input[i] > (132+offset3));

int rt3_0_34 = (input[i] > (137+offset3));
int t3_0_34 = rt3_0_34 * (2 + (input[i] > (138+offset3)));
t3_0_34 += (1 - rt3_0_34) * (input[i] > (136+offset3));

int rt3_0_35 = (input[i] > (141+offset3));
int t3_0_35 = rt3_0_35 * (2 + (input[i] > (142+offset3)));
t3_0_35 += (1 - rt3_0_35) * (input[i] > (140+offset3));

int rt3_0_36 = (input[i] > (145+offset3));
int t3_0_36 = rt3_0_36 * (2 + (input[i] > (146+offset3)));
t3_0_36 += (1 - rt3_0_36) * (input[i] > (144+offset3));

int rt3_0_37 = (input[i] > (149+offset3));
int t3_0_37 = rt3_0_37 * (2 + (input[i] > (150+offset3)));
t3_0_37 += (1 - rt3_0_37) * (input[i] > (148+offset3));

int rt3_0_38 = (input[i] > (153+offset3));
int t3_0_38 = rt3_0_38 * (2 + (input[i] > (154+offset3)));
t3_0_38 += (1 - rt3_0_38) * (input[i] > (152+offset3));

int rt3_0_39 = (input[i] > (157+offset3));
int t3_0_39 = rt3_0_39 * (2 + (input[i] > (158+offset3)));
t3_0_39 += (1 - rt3_0_39) * (input[i] > (156+offset3));

int rt3_0_40 = (input[i] > (161+offset3));
int t3_0_40 = rt3_0_40 * (2 + (input[i] > (162+offset3)));
t3_0_40 += (1 - rt3_0_40) * (input[i] > (160+offset3));

int rt3_0_41 = (input[i] > (165+offset3));
int t3_0_41 = rt3_0_41 * (2 + (input[i] > (166+offset3)));
t3_0_41 += (1 - rt3_0_41) * (input[i] > (164+offset3));

int rt3_0_42 = (input[i] > (169+offset3));
int t3_0_42 = rt3_0_42 * (2 + (input[i] > (170+offset3)));
t3_0_42 += (1 - rt3_0_42) * (input[i] > (168+offset3));

int rt3_0_43 = (input[i] > (173+offset3));
int t3_0_43 = rt3_0_43 * (2 + (input[i] > (174+offset3)));
t3_0_43 += (1 - rt3_0_43) * (input[i] > (172+offset3));

int rt3_0_44 = (input[i] > (177+offset3));
int t3_0_44 = rt3_0_44 * (2 + (input[i] > (178+offset3)));
t3_0_44 += (1 - rt3_0_44) * (input[i] > (176+offset3));

int rt3_0_45 = (input[i] > (181+offset3));
int t3_0_45 = rt3_0_45 * (2 + (input[i] > (182+offset3)));
t3_0_45 += (1 - rt3_0_45) * (input[i] > (180+offset3));

int rt3_0_46 = (input[i] > (185+offset3));
int t3_0_46 = rt3_0_46 * (2 + (input[i] > (186+offset3)));
t3_0_46 += (1 - rt3_0_46) * (input[i] > (184+offset3));

int rt3_0_47 = (input[i] > (189+offset3));
int t3_0_47 = rt3_0_47 * (2 + (input[i] > (190+offset3)));
t3_0_47 += (1 - rt3_0_47) * (input[i] > (188+offset3));

int rt3_0_48 = (input[i] > (193+offset3));
int t3_0_48 = rt3_0_48 * (2 + (input[i] > (194+offset3)));
t3_0_48 += (1 - rt3_0_48) * (input[i] > (192+offset3));

int rt3_0_49 = (input[i] > (197+offset3));
int t3_0_49 = rt3_0_49 * (2 + (input[i] > (198+offset3)));
t3_0_49 += (1 - rt3_0_49) * (input[i] > (196+offset3));

int rt3_0_50 = (input[i] > (201+offset3));
int t3_0_50 = rt3_0_50 * (2 + (input[i] > (202+offset3)));
t3_0_50 += (1 - rt3_0_50) * (input[i] > (200+offset3));

int rt3_0_51 = (input[i] > (205+offset3));
int t3_0_51 = rt3_0_51 * (2 + (input[i] > (206+offset3)));
t3_0_51 += (1 - rt3_0_51) * (input[i] > (204+offset3));

int rt3_0_52 = (input[i] > (209+offset3));
int t3_0_52 = rt3_0_52 * (2 + (input[i] > (210+offset3)));
t3_0_52 += (1 - rt3_0_52) * (input[i] > (208+offset3));

int rt3_0_53 = (input[i] > (213+offset3));
int t3_0_53 = rt3_0_53 * (2 + (input[i] > (214+offset3)));
t3_0_53 += (1 - rt3_0_53) * (input[i] > (212+offset3));

int rt3_0_54 = (input[i] > (217+offset3));
int t3_0_54 = rt3_0_54 * (2 + (input[i] > (218+offset3)));
t3_0_54 += (1 - rt3_0_54) * (input[i] > (216+offset3));

int rt3_0_55 = (input[i] > (221+offset3));
int t3_0_55 = rt3_0_55 * (2 + (input[i] > (222+offset3)));
t3_0_55 += (1 - rt3_0_55) * (input[i] > (220+offset3));

int rt3_0_56 = (input[i] > (225+offset3));
int t3_0_56 = rt3_0_56 * (2 + (input[i] > (226+offset3)));
t3_0_56 += (1 - rt3_0_56) * (input[i] > (224+offset3));

int rt3_0_57 = (input[i] > (229+offset3));
int t3_0_57 = rt3_0_57 * (2 + (input[i] > (230+offset3)));
t3_0_57 += (1 - rt3_0_57) * (input[i] > (228+offset3));

int rt3_0_58 = (input[i] > (233+offset3));
int t3_0_58 = rt3_0_58 * (2 + (input[i] > (234+offset3)));
t3_0_58 += (1 - rt3_0_58) * (input[i] > (232+offset3));

int rt3_0_59 = (input[i] > (237+offset3));
int t3_0_59 = rt3_0_59 * (2 + (input[i] > (238+offset3)));
t3_0_59 += (1 - rt3_0_59) * (input[i] > (236+offset3));

int rt3_0_60 = (input[i] > (241+offset3));
int t3_0_60 = rt3_0_60 * (2 + (input[i] > (242+offset3)));
t3_0_60 += (1 - rt3_0_60) * (input[i] > (240+offset3));

int rt3_0_61 = (input[i] > (245+offset3));
int t3_0_61 = rt3_0_61 * (2 + (input[i] > (246+offset3)));
t3_0_61 += (1 - rt3_0_61) * (input[i] > (244+offset3));

int rt3_0_62 = (input[i] > (249+offset3));
int t3_0_62 = rt3_0_62 * (2 + (input[i] > (250+offset3)));
t3_0_62 += (1 - rt3_0_62) * (input[i] > (248+offset3));

int rt3_0_63 = (input[i] > (253+offset3));
int t3_0_63 = rt3_0_63 * (2 + (input[i] > (254+offset3)));
t3_0_63 += (1 - rt3_0_63) * (input[i] > (252+offset3));

int rt3_1_0 = (input[i] > (3+offset3));
int t3_1_0 = rt3_1_0 * (4 + t3_0_1);
t3_1_0 += (1 - rt3_1_0) * t3_0_0;

int rt3_1_1 = (input[i] > (11+offset3));
int t3_1_1 = rt3_1_1 * (4 + t3_0_3);
t3_1_1 += (1 - rt3_1_1) * t3_0_2;

int rt3_1_2 = (input[i] > (19+offset3));
int t3_1_2 = rt3_1_2 * (4 + t3_0_5);
t3_1_2 += (1 - rt3_1_2) * t3_0_4;

int rt3_1_3 = (input[i] > (27+offset3));
int t3_1_3 = rt3_1_3 * (4 + t3_0_7);
t3_1_3 += (1 - rt3_1_3) * t3_0_6;

int rt3_1_4 = (input[i] > (35+offset3));
int t3_1_4 = rt3_1_4 * (4 + t3_0_9);
t3_1_4 += (1 - rt3_1_4) * t3_0_8;

int rt3_1_5 = (input[i] > (43+offset3));
int t3_1_5 = rt3_1_5 * (4 + t3_0_11);
t3_1_5 += (1 - rt3_1_5) * t3_0_10;

int rt3_1_6 = (input[i] > (51+offset3));
int t3_1_6 = rt3_1_6 * (4 + t3_0_13);
t3_1_6 += (1 - rt3_1_6) * t3_0_12;

int rt3_1_7 = (input[i] > (59+offset3));
int t3_1_7 = rt3_1_7 * (4 + t3_0_15);
t3_1_7 += (1 - rt3_1_7) * t3_0_14;

int rt3_1_8 = (input[i] > (67+offset3));
int t3_1_8 = rt3_1_8 * (4 + t3_0_17);
t3_1_8 += (1 - rt3_1_8) * t3_0_16;

int rt3_1_9 = (input[i] > (75+offset3));
int t3_1_9 = rt3_1_9 * (4 + t3_0_19);
t3_1_9 += (1 - rt3_1_9) * t3_0_18;

int rt3_1_10 = (input[i] > (83+offset3));
int t3_1_10 = rt3_1_10 * (4 + t3_0_21);
t3_1_10 += (1 - rt3_1_10) * t3_0_20;

int rt3_1_11 = (input[i] > (91+offset3));
int t3_1_11 = rt3_1_11 * (4 + t3_0_23);
t3_1_11 += (1 - rt3_1_11) * t3_0_22;

int rt3_1_12 = (input[i] > (99+offset3));
int t3_1_12 = rt3_1_12 * (4 + t3_0_25);
t3_1_12 += (1 - rt3_1_12) * t3_0_24;

int rt3_1_13 = (input[i] > (107+offset3));
int t3_1_13 = rt3_1_13 * (4 + t3_0_27);
t3_1_13 += (1 - rt3_1_13) * t3_0_26;

int rt3_1_14 = (input[i] > (115+offset3));
int t3_1_14 = rt3_1_14 * (4 + t3_0_29);
t3_1_14 += (1 - rt3_1_14) * t3_0_28;

int rt3_1_15 = (input[i] > (123+offset3));
int t3_1_15 = rt3_1_15 * (4 + t3_0_31);
t3_1_15 += (1 - rt3_1_15) * t3_0_30;

int rt3_1_16 = (input[i] > (131+offset3));
int t3_1_16 = rt3_1_16 * (4 + t3_0_33);
t3_1_16 += (1 - rt3_1_16) * t3_0_32;

int rt3_1_17 = (input[i] > (139+offset3));
int t3_1_17 = rt3_1_17 * (4 + t3_0_35);
t3_1_17 += (1 - rt3_1_17) * t3_0_34;

int rt3_1_18 = (input[i] > (147+offset3));
int t3_1_18 = rt3_1_18 * (4 + t3_0_37);
t3_1_18 += (1 - rt3_1_18) * t3_0_36;

int rt3_1_19 = (input[i] > (155+offset3));
int t3_1_19 = rt3_1_19 * (4 + t3_0_39);
t3_1_19 += (1 - rt3_1_19) * t3_0_38;

int rt3_1_20 = (input[i] > (163+offset3));
int t3_1_20 = rt3_1_20 * (4 + t3_0_41);
t3_1_20 += (1 - rt3_1_20) * t3_0_40;

int rt3_1_21 = (input[i] > (171+offset3));
int t3_1_21 = rt3_1_21 * (4 + t3_0_43);
t3_1_21 += (1 - rt3_1_21) * t3_0_42;

int rt3_1_22 = (input[i] > (179+offset3));
int t3_1_22 = rt3_1_22 * (4 + t3_0_45);
t3_1_22 += (1 - rt3_1_22) * t3_0_44;

int rt3_1_23 = (input[i] > (187+offset3));
int t3_1_23 = rt3_1_23 * (4 + t3_0_47);
t3_1_23 += (1 - rt3_1_23) * t3_0_46;

int rt3_1_24 = (input[i] > (195+offset3));
int t3_1_24 = rt3_1_24 * (4 + t3_0_49);
t3_1_24 += (1 - rt3_1_24) * t3_0_48;

int rt3_1_25 = (input[i] > (203+offset3));
int t3_1_25 = rt3_1_25 * (4 + t3_0_51);
t3_1_25 += (1 - rt3_1_25) * t3_0_50;

int rt3_1_26 = (input[i] > (211+offset3));
int t3_1_26 = rt3_1_26 * (4 + t3_0_53);
t3_1_26 += (1 - rt3_1_26) * t3_0_52;

int rt3_1_27 = (input[i] > (219+offset3));
int t3_1_27 = rt3_1_27 * (4 + t3_0_55);
t3_1_27 += (1 - rt3_1_27) * t3_0_54;

int rt3_1_28 = (input[i] > (227+offset3));
int t3_1_28 = rt3_1_28 * (4 + t3_0_57);
t3_1_28 += (1 - rt3_1_28) * t3_0_56;

int rt3_1_29 = (input[i] > (235+offset3));
int t3_1_29 = rt3_1_29 * (4 + t3_0_59);
t3_1_29 += (1 - rt3_1_29) * t3_0_58;

int rt3_1_30 = (input[i] > (243+offset3));
int t3_1_30 = rt3_1_30 * (4 + t3_0_61);
t3_1_30 += (1 - rt3_1_30) * t3_0_60;

int rt3_1_31 = (input[i] > (251+offset3));
int t3_1_31 = rt3_1_31 * (4 + t3_0_63);
t3_1_31 += (1 - rt3_1_31) * t3_0_62;

int rt3_2_0 = (input[i] > (7+offset3));
int t3_2_0 = rt3_2_0 * (8 + t3_1_1);
t3_2_0 += (1 - rt3_2_0) * t3_1_0;

int rt3_2_1 = (input[i] > (23+offset3));
int t3_2_1 = rt3_2_1 * (8 + t3_1_3);
t3_2_1 += (1 - rt3_2_1) * t3_1_2;

int rt3_2_2 = (input[i] > (39+offset3));
int t3_2_2 = rt3_2_2 * (8 + t3_1_5);
t3_2_2 += (1 - rt3_2_2) * t3_1_4;

int rt3_2_3 = (input[i] > (55+offset3));
int t3_2_3 = rt3_2_3 * (8 + t3_1_7);
t3_2_3 += (1 - rt3_2_3) * t3_1_6;

int rt3_2_4 = (input[i] > (71+offset3));
int t3_2_4 = rt3_2_4 * (8 + t3_1_9);
t3_2_4 += (1 - rt3_2_4) * t3_1_8;

int rt3_2_5 = (input[i] > (87+offset3));
int t3_2_5 = rt3_2_5 * (8 + t3_1_11);
t3_2_5 += (1 - rt3_2_5) * t3_1_10;

int rt3_2_6 = (input[i] > (103+offset3));
int t3_2_6 = rt3_2_6 * (8 + t3_1_13);
t3_2_6 += (1 - rt3_2_6) * t3_1_12;

int rt3_2_7 = (input[i] > (119+offset3));
int t3_2_7 = rt3_2_7 * (8 + t3_1_15);
t3_2_7 += (1 - rt3_2_7) * t3_1_14;

int rt3_2_8 = (input[i] > (135+offset3));
int t3_2_8 = rt3_2_8 * (8 + t3_1_17);
t3_2_8 += (1 - rt3_2_8) * t3_1_16;

int rt3_2_9 = (input[i] > (151+offset3));
int t3_2_9 = rt3_2_9 * (8 + t3_1_19);
t3_2_9 += (1 - rt3_2_9) * t3_1_18;

int rt3_2_10 = (input[i] > (167+offset3));
int t3_2_10 = rt3_2_10 * (8 + t3_1_21);
t3_2_10 += (1 - rt3_2_10) * t3_1_20;

int rt3_2_11 = (input[i] > (183+offset3));
int t3_2_11 = rt3_2_11 * (8 + t3_1_23);
t3_2_11 += (1 - rt3_2_11) * t3_1_22;

int rt3_2_12 = (input[i] > (199+offset3));
int t3_2_12 = rt3_2_12 * (8 + t3_1_25);
t3_2_12 += (1 - rt3_2_12) * t3_1_24;

int rt3_2_13 = (input[i] > (215+offset3));
int t3_2_13 = rt3_2_13 * (8 + t3_1_27);
t3_2_13 += (1 - rt3_2_13) * t3_1_26;

int rt3_2_14 = (input[i] > (231+offset3));
int t3_2_14 = rt3_2_14 * (8 + t3_1_29);
t3_2_14 += (1 - rt3_2_14) * t3_1_28;

int rt3_2_15 = (input[i] > (247+offset3));
int t3_2_15 = rt3_2_15 * (8 + t3_1_31);
t3_2_15 += (1 - rt3_2_15) * t3_1_30;

int rt3_3_0 = (input[i] > (15+offset3));
int t3_3_0 = rt3_3_0 * (16 + t3_2_1);
t3_3_0 += (1 - rt3_3_0) * t3_2_0;

int rt3_3_1 = (input[i] > (47+offset3));
int t3_3_1 = rt3_3_1 * (16 + t3_2_3);
t3_3_1 += (1 - rt3_3_1) * t3_2_2;

int rt3_3_2 = (input[i] > (79+offset3));
int t3_3_2 = rt3_3_2 * (16 + t3_2_5);
t3_3_2 += (1 - rt3_3_2) * t3_2_4;

int rt3_3_3 = (input[i] > (111+offset3));
int t3_3_3 = rt3_3_3 * (16 + t3_2_7);
t3_3_3 += (1 - rt3_3_3) * t3_2_6;

int rt3_3_4 = (input[i] > (143+offset3));
int t3_3_4 = rt3_3_4 * (16 + t3_2_9);
t3_3_4 += (1 - rt3_3_4) * t3_2_8;

int rt3_3_5 = (input[i] > (175+offset3));
int t3_3_5 = rt3_3_5 * (16 + t3_2_11);
t3_3_5 += (1 - rt3_3_5) * t3_2_10;

int rt3_3_6 = (input[i] > (207+offset3));
int t3_3_6 = rt3_3_6 * (16 + t3_2_13);
t3_3_6 += (1 - rt3_3_6) * t3_2_12;

int rt3_3_7 = (input[i] > (239+offset3));
int t3_3_7 = rt3_3_7 * (16 + t3_2_15);
t3_3_7 += (1 - rt3_3_7) * t3_2_14;

int rt3_4_0 = (input[i] > (31+offset3));
int t3_4_0 = rt3_4_0 * (32 + t3_3_1);
t3_4_0 += (1 - rt3_4_0) * t3_3_0;

int rt3_4_1 = (input[i] > (95+offset3));
int t3_4_1 = rt3_4_1 * (32 + t3_3_3);
t3_4_1 += (1 - rt3_4_1) * t3_3_2;

int rt3_4_2 = (input[i] > (159+offset3));
int t3_4_2 = rt3_4_2 * (32 + t3_3_5);
t3_4_2 += (1 - rt3_4_2) * t3_3_4;

int rt3_4_3 = (input[i] > (223+offset3));
int t3_4_3 = rt3_4_3 * (32 + t3_3_7);
t3_4_3 += (1 - rt3_4_3) * t3_3_6;

int rt3_5_0 = (input[i] > (63+offset3));
int t3_5_0 = rt3_5_0 * (64 + t3_4_1);
t3_5_0 += (1 - rt3_5_0) * t3_4_0;

int rt3_5_1 = (input[i] > (191+offset3));
int t3_5_1 = rt3_5_1 * (64 + t3_4_3);
t3_5_1 += (1 - rt3_5_1) * t3_4_2;

int rt3_6_0 = (input[i] > (127+offset3));
int t3_6_0 = rt3_6_0 * (128 + t3_5_1);
t3_6_0 += (1 - rt3_6_0) * t3_5_0;

vR[i] = t0_6_0+ (t0_6_0 == t1_6_0)+ (t0_6_0 == t2_6_0)+ (t0_6_0 == t3_6_0);
	}
}

int main(int argc, char **argv)
{
	float elapsed_time;
	// set up device
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("[CUDA - Random Forest]: EXP %s\n", EXP_NAME);
	printf("[CUDA - Random Forest]: Using Device %d: %s\n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));

	// set up data size of vectors
	int nElem = N_ELEM;
	printf("[CUDA - Random Forest]: Vector Size %d\n", nElem);

	// malloc host memory
	size_t nBytes = nElem * sizeof(float);
	float *h_vR;
	h_vR = (float *)malloc(nBytes);
	float *h_input;
	h_input = (float *)malloc(nBytes);

	printf("[CUDA - Random Forest]: Start Reading Dataset\n");
	initData(h_input, nElem);
	printf("[CUDA - Random Forest]: Finished Reading Dataset\n");
	memset(h_vR, 0, nBytes);

	// malloc device global memory
	float *d_vR;
	CHECK(hipMalloc((float **)&d_vR, nBytes));
	float *d_input;
	CHECK(hipMalloc((float **)&d_input, nBytes));

	// transfer data from host to device
	CHECK(hipMemcpy(d_input, h_input, nBytes, hipMemcpyHostToDevice));

	// invoke kernel at host side
	int iLen = 512;
	dim3 block(iLen);
	dim3 grid((nElem + block.x - 1) / block.x);
	hipEvent_t start, stop;
	CHECK(hipEventCreate(&start));
	CHECK(hipEventCreate(&stop));

	// record start event
	CHECK(hipEventRecord(start, 0));
#if defined(_2TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, d_vR);
#elif defined(_3TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, d_vR);
#elif defined(_4TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, d_vR);
#else
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, d_vR);
#endif
	CHECK(hipEventRecord(stop, 0));
	CHECK(hipEventSynchronize(stop));
	hipDeviceSynchronize();

	// calculate elapsed time
	CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
	printf("[CUDA - Random Forest]: %s - execution time = %.6fms\n", EXP_NAME, elapsed_time);
	registerTime(elapsed_time);

	// copy kernel result back to host side
	CHECK(hipMemcpy(h_vR, d_vR, nBytes, hipMemcpyDeviceToHost));
	CHECK(hipGetLastError());

	//Verify the answers
	char flag = 1;
	for (int i = 0; i < nElem; i++)
	{
		if (i < 32)
		{
			printf("%.3f ", h_vR[i]);
		}
		if (h_input[i] != h_vR[i])
		{
			flag = 0;
			break;
		}
	}
	if (flag == 1)
	{
//		printf("Return correct");
	}
	else
	{
//		printf("Error!!");
	}
	printf("\n ");

	// free device memory
	hipFree(d_input);
	hipFree(d_vR);

	// free host memory
	free(h_input);
	free(h_vR);
	return (0);
}

void initData(float *ip, int size)
{
	for (int i = 0; i < size; i++)
	{
		ip[i] = (float)(0);
	}
	return;
}

void registerTime(float value)
{
	printf("[CUDA - Random Forest]: Time spent: %f.\n", value);
}
