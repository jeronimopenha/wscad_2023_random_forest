#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>

#define N_ELEM 100000000

#define WARP_SIZE 32

#define CHECK(call)                                                \
	{                                                              \
		const hipError_t error = call;                            \
		if (error != hipSuccess)                                  \
		{                                                          \
			fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
			fprintf(stderr, "code: %d, reason: %s\n", error,       \
					hipGetErrorString(error));                    \
		}                                                          \
	}

void initData(float *ip, int size);
void registerTime(float value);

#define _MOD 512
#define _1TREE
#define EXP_NAME "_RF_CALC_1TREE_9LV_1INPUT_100000000"

// RF_with_if

__global__ void RF_kernel(float *input, const int n, const int offset0, float *vR)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
int rt0_0_0 = (input[i] > (1+offset0));
int t0_0_0 = rt0_0_0 * (2 + (input[i] > (2+offset0)));
t0_0_0 += (1 - rt0_0_0) * (input[i] > (0+offset0));

int rt0_0_1 = (input[i] > (5+offset0));
int t0_0_1 = rt0_0_1 * (2 + (input[i] > (6+offset0)));
t0_0_1 += (1 - rt0_0_1) * (input[i] > (4+offset0));

int rt0_0_2 = (input[i] > (9+offset0));
int t0_0_2 = rt0_0_2 * (2 + (input[i] > (10+offset0)));
t0_0_2 += (1 - rt0_0_2) * (input[i] > (8+offset0));

int rt0_0_3 = (input[i] > (13+offset0));
int t0_0_3 = rt0_0_3 * (2 + (input[i] > (14+offset0)));
t0_0_3 += (1 - rt0_0_3) * (input[i] > (12+offset0));

int rt0_0_4 = (input[i] > (17+offset0));
int t0_0_4 = rt0_0_4 * (2 + (input[i] > (18+offset0)));
t0_0_4 += (1 - rt0_0_4) * (input[i] > (16+offset0));

int rt0_0_5 = (input[i] > (21+offset0));
int t0_0_5 = rt0_0_5 * (2 + (input[i] > (22+offset0)));
t0_0_5 += (1 - rt0_0_5) * (input[i] > (20+offset0));

int rt0_0_6 = (input[i] > (25+offset0));
int t0_0_6 = rt0_0_6 * (2 + (input[i] > (26+offset0)));
t0_0_6 += (1 - rt0_0_6) * (input[i] > (24+offset0));

int rt0_0_7 = (input[i] > (29+offset0));
int t0_0_7 = rt0_0_7 * (2 + (input[i] > (30+offset0)));
t0_0_7 += (1 - rt0_0_7) * (input[i] > (28+offset0));

int rt0_0_8 = (input[i] > (33+offset0));
int t0_0_8 = rt0_0_8 * (2 + (input[i] > (34+offset0)));
t0_0_8 += (1 - rt0_0_8) * (input[i] > (32+offset0));

int rt0_0_9 = (input[i] > (37+offset0));
int t0_0_9 = rt0_0_9 * (2 + (input[i] > (38+offset0)));
t0_0_9 += (1 - rt0_0_9) * (input[i] > (36+offset0));

int rt0_0_10 = (input[i] > (41+offset0));
int t0_0_10 = rt0_0_10 * (2 + (input[i] > (42+offset0)));
t0_0_10 += (1 - rt0_0_10) * (input[i] > (40+offset0));

int rt0_0_11 = (input[i] > (45+offset0));
int t0_0_11 = rt0_0_11 * (2 + (input[i] > (46+offset0)));
t0_0_11 += (1 - rt0_0_11) * (input[i] > (44+offset0));

int rt0_0_12 = (input[i] > (49+offset0));
int t0_0_12 = rt0_0_12 * (2 + (input[i] > (50+offset0)));
t0_0_12 += (1 - rt0_0_12) * (input[i] > (48+offset0));

int rt0_0_13 = (input[i] > (53+offset0));
int t0_0_13 = rt0_0_13 * (2 + (input[i] > (54+offset0)));
t0_0_13 += (1 - rt0_0_13) * (input[i] > (52+offset0));

int rt0_0_14 = (input[i] > (57+offset0));
int t0_0_14 = rt0_0_14 * (2 + (input[i] > (58+offset0)));
t0_0_14 += (1 - rt0_0_14) * (input[i] > (56+offset0));

int rt0_0_15 = (input[i] > (61+offset0));
int t0_0_15 = rt0_0_15 * (2 + (input[i] > (62+offset0)));
t0_0_15 += (1 - rt0_0_15) * (input[i] > (60+offset0));

int rt0_0_16 = (input[i] > (65+offset0));
int t0_0_16 = rt0_0_16 * (2 + (input[i] > (66+offset0)));
t0_0_16 += (1 - rt0_0_16) * (input[i] > (64+offset0));

int rt0_0_17 = (input[i] > (69+offset0));
int t0_0_17 = rt0_0_17 * (2 + (input[i] > (70+offset0)));
t0_0_17 += (1 - rt0_0_17) * (input[i] > (68+offset0));

int rt0_0_18 = (input[i] > (73+offset0));
int t0_0_18 = rt0_0_18 * (2 + (input[i] > (74+offset0)));
t0_0_18 += (1 - rt0_0_18) * (input[i] > (72+offset0));

int rt0_0_19 = (input[i] > (77+offset0));
int t0_0_19 = rt0_0_19 * (2 + (input[i] > (78+offset0)));
t0_0_19 += (1 - rt0_0_19) * (input[i] > (76+offset0));

int rt0_0_20 = (input[i] > (81+offset0));
int t0_0_20 = rt0_0_20 * (2 + (input[i] > (82+offset0)));
t0_0_20 += (1 - rt0_0_20) * (input[i] > (80+offset0));

int rt0_0_21 = (input[i] > (85+offset0));
int t0_0_21 = rt0_0_21 * (2 + (input[i] > (86+offset0)));
t0_0_21 += (1 - rt0_0_21) * (input[i] > (84+offset0));

int rt0_0_22 = (input[i] > (89+offset0));
int t0_0_22 = rt0_0_22 * (2 + (input[i] > (90+offset0)));
t0_0_22 += (1 - rt0_0_22) * (input[i] > (88+offset0));

int rt0_0_23 = (input[i] > (93+offset0));
int t0_0_23 = rt0_0_23 * (2 + (input[i] > (94+offset0)));
t0_0_23 += (1 - rt0_0_23) * (input[i] > (92+offset0));

int rt0_0_24 = (input[i] > (97+offset0));
int t0_0_24 = rt0_0_24 * (2 + (input[i] > (98+offset0)));
t0_0_24 += (1 - rt0_0_24) * (input[i] > (96+offset0));

int rt0_0_25 = (input[i] > (101+offset0));
int t0_0_25 = rt0_0_25 * (2 + (input[i] > (102+offset0)));
t0_0_25 += (1 - rt0_0_25) * (input[i] > (100+offset0));

int rt0_0_26 = (input[i] > (105+offset0));
int t0_0_26 = rt0_0_26 * (2 + (input[i] > (106+offset0)));
t0_0_26 += (1 - rt0_0_26) * (input[i] > (104+offset0));

int rt0_0_27 = (input[i] > (109+offset0));
int t0_0_27 = rt0_0_27 * (2 + (input[i] > (110+offset0)));
t0_0_27 += (1 - rt0_0_27) * (input[i] > (108+offset0));

int rt0_0_28 = (input[i] > (113+offset0));
int t0_0_28 = rt0_0_28 * (2 + (input[i] > (114+offset0)));
t0_0_28 += (1 - rt0_0_28) * (input[i] > (112+offset0));

int rt0_0_29 = (input[i] > (117+offset0));
int t0_0_29 = rt0_0_29 * (2 + (input[i] > (118+offset0)));
t0_0_29 += (1 - rt0_0_29) * (input[i] > (116+offset0));

int rt0_0_30 = (input[i] > (121+offset0));
int t0_0_30 = rt0_0_30 * (2 + (input[i] > (122+offset0)));
t0_0_30 += (1 - rt0_0_30) * (input[i] > (120+offset0));

int rt0_0_31 = (input[i] > (125+offset0));
int t0_0_31 = rt0_0_31 * (2 + (input[i] > (126+offset0)));
t0_0_31 += (1 - rt0_0_31) * (input[i] > (124+offset0));

int rt0_0_32 = (input[i] > (129+offset0));
int t0_0_32 = rt0_0_32 * (2 + (input[i] > (130+offset0)));
t0_0_32 += (1 - rt0_0_32) * (input[i] > (128+offset0));

int rt0_0_33 = (input[i] > (133+offset0));
int t0_0_33 = rt0_0_33 * (2 + (input[i] > (134+offset0)));
t0_0_33 += (1 - rt0_0_33) * (input[i] > (132+offset0));

int rt0_0_34 = (input[i] > (137+offset0));
int t0_0_34 = rt0_0_34 * (2 + (input[i] > (138+offset0)));
t0_0_34 += (1 - rt0_0_34) * (input[i] > (136+offset0));

int rt0_0_35 = (input[i] > (141+offset0));
int t0_0_35 = rt0_0_35 * (2 + (input[i] > (142+offset0)));
t0_0_35 += (1 - rt0_0_35) * (input[i] > (140+offset0));

int rt0_0_36 = (input[i] > (145+offset0));
int t0_0_36 = rt0_0_36 * (2 + (input[i] > (146+offset0)));
t0_0_36 += (1 - rt0_0_36) * (input[i] > (144+offset0));

int rt0_0_37 = (input[i] > (149+offset0));
int t0_0_37 = rt0_0_37 * (2 + (input[i] > (150+offset0)));
t0_0_37 += (1 - rt0_0_37) * (input[i] > (148+offset0));

int rt0_0_38 = (input[i] > (153+offset0));
int t0_0_38 = rt0_0_38 * (2 + (input[i] > (154+offset0)));
t0_0_38 += (1 - rt0_0_38) * (input[i] > (152+offset0));

int rt0_0_39 = (input[i] > (157+offset0));
int t0_0_39 = rt0_0_39 * (2 + (input[i] > (158+offset0)));
t0_0_39 += (1 - rt0_0_39) * (input[i] > (156+offset0));

int rt0_0_40 = (input[i] > (161+offset0));
int t0_0_40 = rt0_0_40 * (2 + (input[i] > (162+offset0)));
t0_0_40 += (1 - rt0_0_40) * (input[i] > (160+offset0));

int rt0_0_41 = (input[i] > (165+offset0));
int t0_0_41 = rt0_0_41 * (2 + (input[i] > (166+offset0)));
t0_0_41 += (1 - rt0_0_41) * (input[i] > (164+offset0));

int rt0_0_42 = (input[i] > (169+offset0));
int t0_0_42 = rt0_0_42 * (2 + (input[i] > (170+offset0)));
t0_0_42 += (1 - rt0_0_42) * (input[i] > (168+offset0));

int rt0_0_43 = (input[i] > (173+offset0));
int t0_0_43 = rt0_0_43 * (2 + (input[i] > (174+offset0)));
t0_0_43 += (1 - rt0_0_43) * (input[i] > (172+offset0));

int rt0_0_44 = (input[i] > (177+offset0));
int t0_0_44 = rt0_0_44 * (2 + (input[i] > (178+offset0)));
t0_0_44 += (1 - rt0_0_44) * (input[i] > (176+offset0));

int rt0_0_45 = (input[i] > (181+offset0));
int t0_0_45 = rt0_0_45 * (2 + (input[i] > (182+offset0)));
t0_0_45 += (1 - rt0_0_45) * (input[i] > (180+offset0));

int rt0_0_46 = (input[i] > (185+offset0));
int t0_0_46 = rt0_0_46 * (2 + (input[i] > (186+offset0)));
t0_0_46 += (1 - rt0_0_46) * (input[i] > (184+offset0));

int rt0_0_47 = (input[i] > (189+offset0));
int t0_0_47 = rt0_0_47 * (2 + (input[i] > (190+offset0)));
t0_0_47 += (1 - rt0_0_47) * (input[i] > (188+offset0));

int rt0_0_48 = (input[i] > (193+offset0));
int t0_0_48 = rt0_0_48 * (2 + (input[i] > (194+offset0)));
t0_0_48 += (1 - rt0_0_48) * (input[i] > (192+offset0));

int rt0_0_49 = (input[i] > (197+offset0));
int t0_0_49 = rt0_0_49 * (2 + (input[i] > (198+offset0)));
t0_0_49 += (1 - rt0_0_49) * (input[i] > (196+offset0));

int rt0_0_50 = (input[i] > (201+offset0));
int t0_0_50 = rt0_0_50 * (2 + (input[i] > (202+offset0)));
t0_0_50 += (1 - rt0_0_50) * (input[i] > (200+offset0));

int rt0_0_51 = (input[i] > (205+offset0));
int t0_0_51 = rt0_0_51 * (2 + (input[i] > (206+offset0)));
t0_0_51 += (1 - rt0_0_51) * (input[i] > (204+offset0));

int rt0_0_52 = (input[i] > (209+offset0));
int t0_0_52 = rt0_0_52 * (2 + (input[i] > (210+offset0)));
t0_0_52 += (1 - rt0_0_52) * (input[i] > (208+offset0));

int rt0_0_53 = (input[i] > (213+offset0));
int t0_0_53 = rt0_0_53 * (2 + (input[i] > (214+offset0)));
t0_0_53 += (1 - rt0_0_53) * (input[i] > (212+offset0));

int rt0_0_54 = (input[i] > (217+offset0));
int t0_0_54 = rt0_0_54 * (2 + (input[i] > (218+offset0)));
t0_0_54 += (1 - rt0_0_54) * (input[i] > (216+offset0));

int rt0_0_55 = (input[i] > (221+offset0));
int t0_0_55 = rt0_0_55 * (2 + (input[i] > (222+offset0)));
t0_0_55 += (1 - rt0_0_55) * (input[i] > (220+offset0));

int rt0_0_56 = (input[i] > (225+offset0));
int t0_0_56 = rt0_0_56 * (2 + (input[i] > (226+offset0)));
t0_0_56 += (1 - rt0_0_56) * (input[i] > (224+offset0));

int rt0_0_57 = (input[i] > (229+offset0));
int t0_0_57 = rt0_0_57 * (2 + (input[i] > (230+offset0)));
t0_0_57 += (1 - rt0_0_57) * (input[i] > (228+offset0));

int rt0_0_58 = (input[i] > (233+offset0));
int t0_0_58 = rt0_0_58 * (2 + (input[i] > (234+offset0)));
t0_0_58 += (1 - rt0_0_58) * (input[i] > (232+offset0));

int rt0_0_59 = (input[i] > (237+offset0));
int t0_0_59 = rt0_0_59 * (2 + (input[i] > (238+offset0)));
t0_0_59 += (1 - rt0_0_59) * (input[i] > (236+offset0));

int rt0_0_60 = (input[i] > (241+offset0));
int t0_0_60 = rt0_0_60 * (2 + (input[i] > (242+offset0)));
t0_0_60 += (1 - rt0_0_60) * (input[i] > (240+offset0));

int rt0_0_61 = (input[i] > (245+offset0));
int t0_0_61 = rt0_0_61 * (2 + (input[i] > (246+offset0)));
t0_0_61 += (1 - rt0_0_61) * (input[i] > (244+offset0));

int rt0_0_62 = (input[i] > (249+offset0));
int t0_0_62 = rt0_0_62 * (2 + (input[i] > (250+offset0)));
t0_0_62 += (1 - rt0_0_62) * (input[i] > (248+offset0));

int rt0_0_63 = (input[i] > (253+offset0));
int t0_0_63 = rt0_0_63 * (2 + (input[i] > (254+offset0)));
t0_0_63 += (1 - rt0_0_63) * (input[i] > (252+offset0));

int rt0_0_64 = (input[i] > (257+offset0));
int t0_0_64 = rt0_0_64 * (2 + (input[i] > (258+offset0)));
t0_0_64 += (1 - rt0_0_64) * (input[i] > (256+offset0));

int rt0_0_65 = (input[i] > (261+offset0));
int t0_0_65 = rt0_0_65 * (2 + (input[i] > (262+offset0)));
t0_0_65 += (1 - rt0_0_65) * (input[i] > (260+offset0));

int rt0_0_66 = (input[i] > (265+offset0));
int t0_0_66 = rt0_0_66 * (2 + (input[i] > (266+offset0)));
t0_0_66 += (1 - rt0_0_66) * (input[i] > (264+offset0));

int rt0_0_67 = (input[i] > (269+offset0));
int t0_0_67 = rt0_0_67 * (2 + (input[i] > (270+offset0)));
t0_0_67 += (1 - rt0_0_67) * (input[i] > (268+offset0));

int rt0_0_68 = (input[i] > (273+offset0));
int t0_0_68 = rt0_0_68 * (2 + (input[i] > (274+offset0)));
t0_0_68 += (1 - rt0_0_68) * (input[i] > (272+offset0));

int rt0_0_69 = (input[i] > (277+offset0));
int t0_0_69 = rt0_0_69 * (2 + (input[i] > (278+offset0)));
t0_0_69 += (1 - rt0_0_69) * (input[i] > (276+offset0));

int rt0_0_70 = (input[i] > (281+offset0));
int t0_0_70 = rt0_0_70 * (2 + (input[i] > (282+offset0)));
t0_0_70 += (1 - rt0_0_70) * (input[i] > (280+offset0));

int rt0_0_71 = (input[i] > (285+offset0));
int t0_0_71 = rt0_0_71 * (2 + (input[i] > (286+offset0)));
t0_0_71 += (1 - rt0_0_71) * (input[i] > (284+offset0));

int rt0_0_72 = (input[i] > (289+offset0));
int t0_0_72 = rt0_0_72 * (2 + (input[i] > (290+offset0)));
t0_0_72 += (1 - rt0_0_72) * (input[i] > (288+offset0));

int rt0_0_73 = (input[i] > (293+offset0));
int t0_0_73 = rt0_0_73 * (2 + (input[i] > (294+offset0)));
t0_0_73 += (1 - rt0_0_73) * (input[i] > (292+offset0));

int rt0_0_74 = (input[i] > (297+offset0));
int t0_0_74 = rt0_0_74 * (2 + (input[i] > (298+offset0)));
t0_0_74 += (1 - rt0_0_74) * (input[i] > (296+offset0));

int rt0_0_75 = (input[i] > (301+offset0));
int t0_0_75 = rt0_0_75 * (2 + (input[i] > (302+offset0)));
t0_0_75 += (1 - rt0_0_75) * (input[i] > (300+offset0));

int rt0_0_76 = (input[i] > (305+offset0));
int t0_0_76 = rt0_0_76 * (2 + (input[i] > (306+offset0)));
t0_0_76 += (1 - rt0_0_76) * (input[i] > (304+offset0));

int rt0_0_77 = (input[i] > (309+offset0));
int t0_0_77 = rt0_0_77 * (2 + (input[i] > (310+offset0)));
t0_0_77 += (1 - rt0_0_77) * (input[i] > (308+offset0));

int rt0_0_78 = (input[i] > (313+offset0));
int t0_0_78 = rt0_0_78 * (2 + (input[i] > (314+offset0)));
t0_0_78 += (1 - rt0_0_78) * (input[i] > (312+offset0));

int rt0_0_79 = (input[i] > (317+offset0));
int t0_0_79 = rt0_0_79 * (2 + (input[i] > (318+offset0)));
t0_0_79 += (1 - rt0_0_79) * (input[i] > (316+offset0));

int rt0_0_80 = (input[i] > (321+offset0));
int t0_0_80 = rt0_0_80 * (2 + (input[i] > (322+offset0)));
t0_0_80 += (1 - rt0_0_80) * (input[i] > (320+offset0));

int rt0_0_81 = (input[i] > (325+offset0));
int t0_0_81 = rt0_0_81 * (2 + (input[i] > (326+offset0)));
t0_0_81 += (1 - rt0_0_81) * (input[i] > (324+offset0));

int rt0_0_82 = (input[i] > (329+offset0));
int t0_0_82 = rt0_0_82 * (2 + (input[i] > (330+offset0)));
t0_0_82 += (1 - rt0_0_82) * (input[i] > (328+offset0));

int rt0_0_83 = (input[i] > (333+offset0));
int t0_0_83 = rt0_0_83 * (2 + (input[i] > (334+offset0)));
t0_0_83 += (1 - rt0_0_83) * (input[i] > (332+offset0));

int rt0_0_84 = (input[i] > (337+offset0));
int t0_0_84 = rt0_0_84 * (2 + (input[i] > (338+offset0)));
t0_0_84 += (1 - rt0_0_84) * (input[i] > (336+offset0));

int rt0_0_85 = (input[i] > (341+offset0));
int t0_0_85 = rt0_0_85 * (2 + (input[i] > (342+offset0)));
t0_0_85 += (1 - rt0_0_85) * (input[i] > (340+offset0));

int rt0_0_86 = (input[i] > (345+offset0));
int t0_0_86 = rt0_0_86 * (2 + (input[i] > (346+offset0)));
t0_0_86 += (1 - rt0_0_86) * (input[i] > (344+offset0));

int rt0_0_87 = (input[i] > (349+offset0));
int t0_0_87 = rt0_0_87 * (2 + (input[i] > (350+offset0)));
t0_0_87 += (1 - rt0_0_87) * (input[i] > (348+offset0));

int rt0_0_88 = (input[i] > (353+offset0));
int t0_0_88 = rt0_0_88 * (2 + (input[i] > (354+offset0)));
t0_0_88 += (1 - rt0_0_88) * (input[i] > (352+offset0));

int rt0_0_89 = (input[i] > (357+offset0));
int t0_0_89 = rt0_0_89 * (2 + (input[i] > (358+offset0)));
t0_0_89 += (1 - rt0_0_89) * (input[i] > (356+offset0));

int rt0_0_90 = (input[i] > (361+offset0));
int t0_0_90 = rt0_0_90 * (2 + (input[i] > (362+offset0)));
t0_0_90 += (1 - rt0_0_90) * (input[i] > (360+offset0));

int rt0_0_91 = (input[i] > (365+offset0));
int t0_0_91 = rt0_0_91 * (2 + (input[i] > (366+offset0)));
t0_0_91 += (1 - rt0_0_91) * (input[i] > (364+offset0));

int rt0_0_92 = (input[i] > (369+offset0));
int t0_0_92 = rt0_0_92 * (2 + (input[i] > (370+offset0)));
t0_0_92 += (1 - rt0_0_92) * (input[i] > (368+offset0));

int rt0_0_93 = (input[i] > (373+offset0));
int t0_0_93 = rt0_0_93 * (2 + (input[i] > (374+offset0)));
t0_0_93 += (1 - rt0_0_93) * (input[i] > (372+offset0));

int rt0_0_94 = (input[i] > (377+offset0));
int t0_0_94 = rt0_0_94 * (2 + (input[i] > (378+offset0)));
t0_0_94 += (1 - rt0_0_94) * (input[i] > (376+offset0));

int rt0_0_95 = (input[i] > (381+offset0));
int t0_0_95 = rt0_0_95 * (2 + (input[i] > (382+offset0)));
t0_0_95 += (1 - rt0_0_95) * (input[i] > (380+offset0));

int rt0_0_96 = (input[i] > (385+offset0));
int t0_0_96 = rt0_0_96 * (2 + (input[i] > (386+offset0)));
t0_0_96 += (1 - rt0_0_96) * (input[i] > (384+offset0));

int rt0_0_97 = (input[i] > (389+offset0));
int t0_0_97 = rt0_0_97 * (2 + (input[i] > (390+offset0)));
t0_0_97 += (1 - rt0_0_97) * (input[i] > (388+offset0));

int rt0_0_98 = (input[i] > (393+offset0));
int t0_0_98 = rt0_0_98 * (2 + (input[i] > (394+offset0)));
t0_0_98 += (1 - rt0_0_98) * (input[i] > (392+offset0));

int rt0_0_99 = (input[i] > (397+offset0));
int t0_0_99 = rt0_0_99 * (2 + (input[i] > (398+offset0)));
t0_0_99 += (1 - rt0_0_99) * (input[i] > (396+offset0));

int rt0_0_100 = (input[i] > (401+offset0));
int t0_0_100 = rt0_0_100 * (2 + (input[i] > (402+offset0)));
t0_0_100 += (1 - rt0_0_100) * (input[i] > (400+offset0));

int rt0_0_101 = (input[i] > (405+offset0));
int t0_0_101 = rt0_0_101 * (2 + (input[i] > (406+offset0)));
t0_0_101 += (1 - rt0_0_101) * (input[i] > (404+offset0));

int rt0_0_102 = (input[i] > (409+offset0));
int t0_0_102 = rt0_0_102 * (2 + (input[i] > (410+offset0)));
t0_0_102 += (1 - rt0_0_102) * (input[i] > (408+offset0));

int rt0_0_103 = (input[i] > (413+offset0));
int t0_0_103 = rt0_0_103 * (2 + (input[i] > (414+offset0)));
t0_0_103 += (1 - rt0_0_103) * (input[i] > (412+offset0));

int rt0_0_104 = (input[i] > (417+offset0));
int t0_0_104 = rt0_0_104 * (2 + (input[i] > (418+offset0)));
t0_0_104 += (1 - rt0_0_104) * (input[i] > (416+offset0));

int rt0_0_105 = (input[i] > (421+offset0));
int t0_0_105 = rt0_0_105 * (2 + (input[i] > (422+offset0)));
t0_0_105 += (1 - rt0_0_105) * (input[i] > (420+offset0));

int rt0_0_106 = (input[i] > (425+offset0));
int t0_0_106 = rt0_0_106 * (2 + (input[i] > (426+offset0)));
t0_0_106 += (1 - rt0_0_106) * (input[i] > (424+offset0));

int rt0_0_107 = (input[i] > (429+offset0));
int t0_0_107 = rt0_0_107 * (2 + (input[i] > (430+offset0)));
t0_0_107 += (1 - rt0_0_107) * (input[i] > (428+offset0));

int rt0_0_108 = (input[i] > (433+offset0));
int t0_0_108 = rt0_0_108 * (2 + (input[i] > (434+offset0)));
t0_0_108 += (1 - rt0_0_108) * (input[i] > (432+offset0));

int rt0_0_109 = (input[i] > (437+offset0));
int t0_0_109 = rt0_0_109 * (2 + (input[i] > (438+offset0)));
t0_0_109 += (1 - rt0_0_109) * (input[i] > (436+offset0));

int rt0_0_110 = (input[i] > (441+offset0));
int t0_0_110 = rt0_0_110 * (2 + (input[i] > (442+offset0)));
t0_0_110 += (1 - rt0_0_110) * (input[i] > (440+offset0));

int rt0_0_111 = (input[i] > (445+offset0));
int t0_0_111 = rt0_0_111 * (2 + (input[i] > (446+offset0)));
t0_0_111 += (1 - rt0_0_111) * (input[i] > (444+offset0));

int rt0_0_112 = (input[i] > (449+offset0));
int t0_0_112 = rt0_0_112 * (2 + (input[i] > (450+offset0)));
t0_0_112 += (1 - rt0_0_112) * (input[i] > (448+offset0));

int rt0_0_113 = (input[i] > (453+offset0));
int t0_0_113 = rt0_0_113 * (2 + (input[i] > (454+offset0)));
t0_0_113 += (1 - rt0_0_113) * (input[i] > (452+offset0));

int rt0_0_114 = (input[i] > (457+offset0));
int t0_0_114 = rt0_0_114 * (2 + (input[i] > (458+offset0)));
t0_0_114 += (1 - rt0_0_114) * (input[i] > (456+offset0));

int rt0_0_115 = (input[i] > (461+offset0));
int t0_0_115 = rt0_0_115 * (2 + (input[i] > (462+offset0)));
t0_0_115 += (1 - rt0_0_115) * (input[i] > (460+offset0));

int rt0_0_116 = (input[i] > (465+offset0));
int t0_0_116 = rt0_0_116 * (2 + (input[i] > (466+offset0)));
t0_0_116 += (1 - rt0_0_116) * (input[i] > (464+offset0));

int rt0_0_117 = (input[i] > (469+offset0));
int t0_0_117 = rt0_0_117 * (2 + (input[i] > (470+offset0)));
t0_0_117 += (1 - rt0_0_117) * (input[i] > (468+offset0));

int rt0_0_118 = (input[i] > (473+offset0));
int t0_0_118 = rt0_0_118 * (2 + (input[i] > (474+offset0)));
t0_0_118 += (1 - rt0_0_118) * (input[i] > (472+offset0));

int rt0_0_119 = (input[i] > (477+offset0));
int t0_0_119 = rt0_0_119 * (2 + (input[i] > (478+offset0)));
t0_0_119 += (1 - rt0_0_119) * (input[i] > (476+offset0));

int rt0_0_120 = (input[i] > (481+offset0));
int t0_0_120 = rt0_0_120 * (2 + (input[i] > (482+offset0)));
t0_0_120 += (1 - rt0_0_120) * (input[i] > (480+offset0));

int rt0_0_121 = (input[i] > (485+offset0));
int t0_0_121 = rt0_0_121 * (2 + (input[i] > (486+offset0)));
t0_0_121 += (1 - rt0_0_121) * (input[i] > (484+offset0));

int rt0_0_122 = (input[i] > (489+offset0));
int t0_0_122 = rt0_0_122 * (2 + (input[i] > (490+offset0)));
t0_0_122 += (1 - rt0_0_122) * (input[i] > (488+offset0));

int rt0_0_123 = (input[i] > (493+offset0));
int t0_0_123 = rt0_0_123 * (2 + (input[i] > (494+offset0)));
t0_0_123 += (1 - rt0_0_123) * (input[i] > (492+offset0));

int rt0_0_124 = (input[i] > (497+offset0));
int t0_0_124 = rt0_0_124 * (2 + (input[i] > (498+offset0)));
t0_0_124 += (1 - rt0_0_124) * (input[i] > (496+offset0));

int rt0_0_125 = (input[i] > (501+offset0));
int t0_0_125 = rt0_0_125 * (2 + (input[i] > (502+offset0)));
t0_0_125 += (1 - rt0_0_125) * (input[i] > (500+offset0));

int rt0_0_126 = (input[i] > (505+offset0));
int t0_0_126 = rt0_0_126 * (2 + (input[i] > (506+offset0)));
t0_0_126 += (1 - rt0_0_126) * (input[i] > (504+offset0));

int rt0_0_127 = (input[i] > (509+offset0));
int t0_0_127 = rt0_0_127 * (2 + (input[i] > (510+offset0)));
t0_0_127 += (1 - rt0_0_127) * (input[i] > (508+offset0));

int rt0_1_0 = (input[i] > (3+offset0));
int t0_1_0 = rt0_1_0 * (4 + t0_0_1);
t0_1_0 += (1 - rt0_1_0) * t0_0_0;

int rt0_1_1 = (input[i] > (11+offset0));
int t0_1_1 = rt0_1_1 * (4 + t0_0_3);
t0_1_1 += (1 - rt0_1_1) * t0_0_2;

int rt0_1_2 = (input[i] > (19+offset0));
int t0_1_2 = rt0_1_2 * (4 + t0_0_5);
t0_1_2 += (1 - rt0_1_2) * t0_0_4;

int rt0_1_3 = (input[i] > (27+offset0));
int t0_1_3 = rt0_1_3 * (4 + t0_0_7);
t0_1_3 += (1 - rt0_1_3) * t0_0_6;

int rt0_1_4 = (input[i] > (35+offset0));
int t0_1_4 = rt0_1_4 * (4 + t0_0_9);
t0_1_4 += (1 - rt0_1_4) * t0_0_8;

int rt0_1_5 = (input[i] > (43+offset0));
int t0_1_5 = rt0_1_5 * (4 + t0_0_11);
t0_1_5 += (1 - rt0_1_5) * t0_0_10;

int rt0_1_6 = (input[i] > (51+offset0));
int t0_1_6 = rt0_1_6 * (4 + t0_0_13);
t0_1_6 += (1 - rt0_1_6) * t0_0_12;

int rt0_1_7 = (input[i] > (59+offset0));
int t0_1_7 = rt0_1_7 * (4 + t0_0_15);
t0_1_7 += (1 - rt0_1_7) * t0_0_14;

int rt0_1_8 = (input[i] > (67+offset0));
int t0_1_8 = rt0_1_8 * (4 + t0_0_17);
t0_1_8 += (1 - rt0_1_8) * t0_0_16;

int rt0_1_9 = (input[i] > (75+offset0));
int t0_1_9 = rt0_1_9 * (4 + t0_0_19);
t0_1_9 += (1 - rt0_1_9) * t0_0_18;

int rt0_1_10 = (input[i] > (83+offset0));
int t0_1_10 = rt0_1_10 * (4 + t0_0_21);
t0_1_10 += (1 - rt0_1_10) * t0_0_20;

int rt0_1_11 = (input[i] > (91+offset0));
int t0_1_11 = rt0_1_11 * (4 + t0_0_23);
t0_1_11 += (1 - rt0_1_11) * t0_0_22;

int rt0_1_12 = (input[i] > (99+offset0));
int t0_1_12 = rt0_1_12 * (4 + t0_0_25);
t0_1_12 += (1 - rt0_1_12) * t0_0_24;

int rt0_1_13 = (input[i] > (107+offset0));
int t0_1_13 = rt0_1_13 * (4 + t0_0_27);
t0_1_13 += (1 - rt0_1_13) * t0_0_26;

int rt0_1_14 = (input[i] > (115+offset0));
int t0_1_14 = rt0_1_14 * (4 + t0_0_29);
t0_1_14 += (1 - rt0_1_14) * t0_0_28;

int rt0_1_15 = (input[i] > (123+offset0));
int t0_1_15 = rt0_1_15 * (4 + t0_0_31);
t0_1_15 += (1 - rt0_1_15) * t0_0_30;

int rt0_1_16 = (input[i] > (131+offset0));
int t0_1_16 = rt0_1_16 * (4 + t0_0_33);
t0_1_16 += (1 - rt0_1_16) * t0_0_32;

int rt0_1_17 = (input[i] > (139+offset0));
int t0_1_17 = rt0_1_17 * (4 + t0_0_35);
t0_1_17 += (1 - rt0_1_17) * t0_0_34;

int rt0_1_18 = (input[i] > (147+offset0));
int t0_1_18 = rt0_1_18 * (4 + t0_0_37);
t0_1_18 += (1 - rt0_1_18) * t0_0_36;

int rt0_1_19 = (input[i] > (155+offset0));
int t0_1_19 = rt0_1_19 * (4 + t0_0_39);
t0_1_19 += (1 - rt0_1_19) * t0_0_38;

int rt0_1_20 = (input[i] > (163+offset0));
int t0_1_20 = rt0_1_20 * (4 + t0_0_41);
t0_1_20 += (1 - rt0_1_20) * t0_0_40;

int rt0_1_21 = (input[i] > (171+offset0));
int t0_1_21 = rt0_1_21 * (4 + t0_0_43);
t0_1_21 += (1 - rt0_1_21) * t0_0_42;

int rt0_1_22 = (input[i] > (179+offset0));
int t0_1_22 = rt0_1_22 * (4 + t0_0_45);
t0_1_22 += (1 - rt0_1_22) * t0_0_44;

int rt0_1_23 = (input[i] > (187+offset0));
int t0_1_23 = rt0_1_23 * (4 + t0_0_47);
t0_1_23 += (1 - rt0_1_23) * t0_0_46;

int rt0_1_24 = (input[i] > (195+offset0));
int t0_1_24 = rt0_1_24 * (4 + t0_0_49);
t0_1_24 += (1 - rt0_1_24) * t0_0_48;

int rt0_1_25 = (input[i] > (203+offset0));
int t0_1_25 = rt0_1_25 * (4 + t0_0_51);
t0_1_25 += (1 - rt0_1_25) * t0_0_50;

int rt0_1_26 = (input[i] > (211+offset0));
int t0_1_26 = rt0_1_26 * (4 + t0_0_53);
t0_1_26 += (1 - rt0_1_26) * t0_0_52;

int rt0_1_27 = (input[i] > (219+offset0));
int t0_1_27 = rt0_1_27 * (4 + t0_0_55);
t0_1_27 += (1 - rt0_1_27) * t0_0_54;

int rt0_1_28 = (input[i] > (227+offset0));
int t0_1_28 = rt0_1_28 * (4 + t0_0_57);
t0_1_28 += (1 - rt0_1_28) * t0_0_56;

int rt0_1_29 = (input[i] > (235+offset0));
int t0_1_29 = rt0_1_29 * (4 + t0_0_59);
t0_1_29 += (1 - rt0_1_29) * t0_0_58;

int rt0_1_30 = (input[i] > (243+offset0));
int t0_1_30 = rt0_1_30 * (4 + t0_0_61);
t0_1_30 += (1 - rt0_1_30) * t0_0_60;

int rt0_1_31 = (input[i] > (251+offset0));
int t0_1_31 = rt0_1_31 * (4 + t0_0_63);
t0_1_31 += (1 - rt0_1_31) * t0_0_62;

int rt0_1_32 = (input[i] > (259+offset0));
int t0_1_32 = rt0_1_32 * (4 + t0_0_65);
t0_1_32 += (1 - rt0_1_32) * t0_0_64;

int rt0_1_33 = (input[i] > (267+offset0));
int t0_1_33 = rt0_1_33 * (4 + t0_0_67);
t0_1_33 += (1 - rt0_1_33) * t0_0_66;

int rt0_1_34 = (input[i] > (275+offset0));
int t0_1_34 = rt0_1_34 * (4 + t0_0_69);
t0_1_34 += (1 - rt0_1_34) * t0_0_68;

int rt0_1_35 = (input[i] > (283+offset0));
int t0_1_35 = rt0_1_35 * (4 + t0_0_71);
t0_1_35 += (1 - rt0_1_35) * t0_0_70;

int rt0_1_36 = (input[i] > (291+offset0));
int t0_1_36 = rt0_1_36 * (4 + t0_0_73);
t0_1_36 += (1 - rt0_1_36) * t0_0_72;

int rt0_1_37 = (input[i] > (299+offset0));
int t0_1_37 = rt0_1_37 * (4 + t0_0_75);
t0_1_37 += (1 - rt0_1_37) * t0_0_74;

int rt0_1_38 = (input[i] > (307+offset0));
int t0_1_38 = rt0_1_38 * (4 + t0_0_77);
t0_1_38 += (1 - rt0_1_38) * t0_0_76;

int rt0_1_39 = (input[i] > (315+offset0));
int t0_1_39 = rt0_1_39 * (4 + t0_0_79);
t0_1_39 += (1 - rt0_1_39) * t0_0_78;

int rt0_1_40 = (input[i] > (323+offset0));
int t0_1_40 = rt0_1_40 * (4 + t0_0_81);
t0_1_40 += (1 - rt0_1_40) * t0_0_80;

int rt0_1_41 = (input[i] > (331+offset0));
int t0_1_41 = rt0_1_41 * (4 + t0_0_83);
t0_1_41 += (1 - rt0_1_41) * t0_0_82;

int rt0_1_42 = (input[i] > (339+offset0));
int t0_1_42 = rt0_1_42 * (4 + t0_0_85);
t0_1_42 += (1 - rt0_1_42) * t0_0_84;

int rt0_1_43 = (input[i] > (347+offset0));
int t0_1_43 = rt0_1_43 * (4 + t0_0_87);
t0_1_43 += (1 - rt0_1_43) * t0_0_86;

int rt0_1_44 = (input[i] > (355+offset0));
int t0_1_44 = rt0_1_44 * (4 + t0_0_89);
t0_1_44 += (1 - rt0_1_44) * t0_0_88;

int rt0_1_45 = (input[i] > (363+offset0));
int t0_1_45 = rt0_1_45 * (4 + t0_0_91);
t0_1_45 += (1 - rt0_1_45) * t0_0_90;

int rt0_1_46 = (input[i] > (371+offset0));
int t0_1_46 = rt0_1_46 * (4 + t0_0_93);
t0_1_46 += (1 - rt0_1_46) * t0_0_92;

int rt0_1_47 = (input[i] > (379+offset0));
int t0_1_47 = rt0_1_47 * (4 + t0_0_95);
t0_1_47 += (1 - rt0_1_47) * t0_0_94;

int rt0_1_48 = (input[i] > (387+offset0));
int t0_1_48 = rt0_1_48 * (4 + t0_0_97);
t0_1_48 += (1 - rt0_1_48) * t0_0_96;

int rt0_1_49 = (input[i] > (395+offset0));
int t0_1_49 = rt0_1_49 * (4 + t0_0_99);
t0_1_49 += (1 - rt0_1_49) * t0_0_98;

int rt0_1_50 = (input[i] > (403+offset0));
int t0_1_50 = rt0_1_50 * (4 + t0_0_101);
t0_1_50 += (1 - rt0_1_50) * t0_0_100;

int rt0_1_51 = (input[i] > (411+offset0));
int t0_1_51 = rt0_1_51 * (4 + t0_0_103);
t0_1_51 += (1 - rt0_1_51) * t0_0_102;

int rt0_1_52 = (input[i] > (419+offset0));
int t0_1_52 = rt0_1_52 * (4 + t0_0_105);
t0_1_52 += (1 - rt0_1_52) * t0_0_104;

int rt0_1_53 = (input[i] > (427+offset0));
int t0_1_53 = rt0_1_53 * (4 + t0_0_107);
t0_1_53 += (1 - rt0_1_53) * t0_0_106;

int rt0_1_54 = (input[i] > (435+offset0));
int t0_1_54 = rt0_1_54 * (4 + t0_0_109);
t0_1_54 += (1 - rt0_1_54) * t0_0_108;

int rt0_1_55 = (input[i] > (443+offset0));
int t0_1_55 = rt0_1_55 * (4 + t0_0_111);
t0_1_55 += (1 - rt0_1_55) * t0_0_110;

int rt0_1_56 = (input[i] > (451+offset0));
int t0_1_56 = rt0_1_56 * (4 + t0_0_113);
t0_1_56 += (1 - rt0_1_56) * t0_0_112;

int rt0_1_57 = (input[i] > (459+offset0));
int t0_1_57 = rt0_1_57 * (4 + t0_0_115);
t0_1_57 += (1 - rt0_1_57) * t0_0_114;

int rt0_1_58 = (input[i] > (467+offset0));
int t0_1_58 = rt0_1_58 * (4 + t0_0_117);
t0_1_58 += (1 - rt0_1_58) * t0_0_116;

int rt0_1_59 = (input[i] > (475+offset0));
int t0_1_59 = rt0_1_59 * (4 + t0_0_119);
t0_1_59 += (1 - rt0_1_59) * t0_0_118;

int rt0_1_60 = (input[i] > (483+offset0));
int t0_1_60 = rt0_1_60 * (4 + t0_0_121);
t0_1_60 += (1 - rt0_1_60) * t0_0_120;

int rt0_1_61 = (input[i] > (491+offset0));
int t0_1_61 = rt0_1_61 * (4 + t0_0_123);
t0_1_61 += (1 - rt0_1_61) * t0_0_122;

int rt0_1_62 = (input[i] > (499+offset0));
int t0_1_62 = rt0_1_62 * (4 + t0_0_125);
t0_1_62 += (1 - rt0_1_62) * t0_0_124;

int rt0_1_63 = (input[i] > (507+offset0));
int t0_1_63 = rt0_1_63 * (4 + t0_0_127);
t0_1_63 += (1 - rt0_1_63) * t0_0_126;

int rt0_2_0 = (input[i] > (7+offset0));
int t0_2_0 = rt0_2_0 * (8 + t0_1_1);
t0_2_0 += (1 - rt0_2_0) * t0_1_0;

int rt0_2_1 = (input[i] > (23+offset0));
int t0_2_1 = rt0_2_1 * (8 + t0_1_3);
t0_2_1 += (1 - rt0_2_1) * t0_1_2;

int rt0_2_2 = (input[i] > (39+offset0));
int t0_2_2 = rt0_2_2 * (8 + t0_1_5);
t0_2_2 += (1 - rt0_2_2) * t0_1_4;

int rt0_2_3 = (input[i] > (55+offset0));
int t0_2_3 = rt0_2_3 * (8 + t0_1_7);
t0_2_3 += (1 - rt0_2_3) * t0_1_6;

int rt0_2_4 = (input[i] > (71+offset0));
int t0_2_4 = rt0_2_4 * (8 + t0_1_9);
t0_2_4 += (1 - rt0_2_4) * t0_1_8;

int rt0_2_5 = (input[i] > (87+offset0));
int t0_2_5 = rt0_2_5 * (8 + t0_1_11);
t0_2_5 += (1 - rt0_2_5) * t0_1_10;

int rt0_2_6 = (input[i] > (103+offset0));
int t0_2_6 = rt0_2_6 * (8 + t0_1_13);
t0_2_6 += (1 - rt0_2_6) * t0_1_12;

int rt0_2_7 = (input[i] > (119+offset0));
int t0_2_7 = rt0_2_7 * (8 + t0_1_15);
t0_2_7 += (1 - rt0_2_7) * t0_1_14;

int rt0_2_8 = (input[i] > (135+offset0));
int t0_2_8 = rt0_2_8 * (8 + t0_1_17);
t0_2_8 += (1 - rt0_2_8) * t0_1_16;

int rt0_2_9 = (input[i] > (151+offset0));
int t0_2_9 = rt0_2_9 * (8 + t0_1_19);
t0_2_9 += (1 - rt0_2_9) * t0_1_18;

int rt0_2_10 = (input[i] > (167+offset0));
int t0_2_10 = rt0_2_10 * (8 + t0_1_21);
t0_2_10 += (1 - rt0_2_10) * t0_1_20;

int rt0_2_11 = (input[i] > (183+offset0));
int t0_2_11 = rt0_2_11 * (8 + t0_1_23);
t0_2_11 += (1 - rt0_2_11) * t0_1_22;

int rt0_2_12 = (input[i] > (199+offset0));
int t0_2_12 = rt0_2_12 * (8 + t0_1_25);
t0_2_12 += (1 - rt0_2_12) * t0_1_24;

int rt0_2_13 = (input[i] > (215+offset0));
int t0_2_13 = rt0_2_13 * (8 + t0_1_27);
t0_2_13 += (1 - rt0_2_13) * t0_1_26;

int rt0_2_14 = (input[i] > (231+offset0));
int t0_2_14 = rt0_2_14 * (8 + t0_1_29);
t0_2_14 += (1 - rt0_2_14) * t0_1_28;

int rt0_2_15 = (input[i] > (247+offset0));
int t0_2_15 = rt0_2_15 * (8 + t0_1_31);
t0_2_15 += (1 - rt0_2_15) * t0_1_30;

int rt0_2_16 = (input[i] > (263+offset0));
int t0_2_16 = rt0_2_16 * (8 + t0_1_33);
t0_2_16 += (1 - rt0_2_16) * t0_1_32;

int rt0_2_17 = (input[i] > (279+offset0));
int t0_2_17 = rt0_2_17 * (8 + t0_1_35);
t0_2_17 += (1 - rt0_2_17) * t0_1_34;

int rt0_2_18 = (input[i] > (295+offset0));
int t0_2_18 = rt0_2_18 * (8 + t0_1_37);
t0_2_18 += (1 - rt0_2_18) * t0_1_36;

int rt0_2_19 = (input[i] > (311+offset0));
int t0_2_19 = rt0_2_19 * (8 + t0_1_39);
t0_2_19 += (1 - rt0_2_19) * t0_1_38;

int rt0_2_20 = (input[i] > (327+offset0));
int t0_2_20 = rt0_2_20 * (8 + t0_1_41);
t0_2_20 += (1 - rt0_2_20) * t0_1_40;

int rt0_2_21 = (input[i] > (343+offset0));
int t0_2_21 = rt0_2_21 * (8 + t0_1_43);
t0_2_21 += (1 - rt0_2_21) * t0_1_42;

int rt0_2_22 = (input[i] > (359+offset0));
int t0_2_22 = rt0_2_22 * (8 + t0_1_45);
t0_2_22 += (1 - rt0_2_22) * t0_1_44;

int rt0_2_23 = (input[i] > (375+offset0));
int t0_2_23 = rt0_2_23 * (8 + t0_1_47);
t0_2_23 += (1 - rt0_2_23) * t0_1_46;

int rt0_2_24 = (input[i] > (391+offset0));
int t0_2_24 = rt0_2_24 * (8 + t0_1_49);
t0_2_24 += (1 - rt0_2_24) * t0_1_48;

int rt0_2_25 = (input[i] > (407+offset0));
int t0_2_25 = rt0_2_25 * (8 + t0_1_51);
t0_2_25 += (1 - rt0_2_25) * t0_1_50;

int rt0_2_26 = (input[i] > (423+offset0));
int t0_2_26 = rt0_2_26 * (8 + t0_1_53);
t0_2_26 += (1 - rt0_2_26) * t0_1_52;

int rt0_2_27 = (input[i] > (439+offset0));
int t0_2_27 = rt0_2_27 * (8 + t0_1_55);
t0_2_27 += (1 - rt0_2_27) * t0_1_54;

int rt0_2_28 = (input[i] > (455+offset0));
int t0_2_28 = rt0_2_28 * (8 + t0_1_57);
t0_2_28 += (1 - rt0_2_28) * t0_1_56;

int rt0_2_29 = (input[i] > (471+offset0));
int t0_2_29 = rt0_2_29 * (8 + t0_1_59);
t0_2_29 += (1 - rt0_2_29) * t0_1_58;

int rt0_2_30 = (input[i] > (487+offset0));
int t0_2_30 = rt0_2_30 * (8 + t0_1_61);
t0_2_30 += (1 - rt0_2_30) * t0_1_60;

int rt0_2_31 = (input[i] > (503+offset0));
int t0_2_31 = rt0_2_31 * (8 + t0_1_63);
t0_2_31 += (1 - rt0_2_31) * t0_1_62;

int rt0_3_0 = (input[i] > (15+offset0));
int t0_3_0 = rt0_3_0 * (16 + t0_2_1);
t0_3_0 += (1 - rt0_3_0) * t0_2_0;

int rt0_3_1 = (input[i] > (47+offset0));
int t0_3_1 = rt0_3_1 * (16 + t0_2_3);
t0_3_1 += (1 - rt0_3_1) * t0_2_2;

int rt0_3_2 = (input[i] > (79+offset0));
int t0_3_2 = rt0_3_2 * (16 + t0_2_5);
t0_3_2 += (1 - rt0_3_2) * t0_2_4;

int rt0_3_3 = (input[i] > (111+offset0));
int t0_3_3 = rt0_3_3 * (16 + t0_2_7);
t0_3_3 += (1 - rt0_3_3) * t0_2_6;

int rt0_3_4 = (input[i] > (143+offset0));
int t0_3_4 = rt0_3_4 * (16 + t0_2_9);
t0_3_4 += (1 - rt0_3_4) * t0_2_8;

int rt0_3_5 = (input[i] > (175+offset0));
int t0_3_5 = rt0_3_5 * (16 + t0_2_11);
t0_3_5 += (1 - rt0_3_5) * t0_2_10;

int rt0_3_6 = (input[i] > (207+offset0));
int t0_3_6 = rt0_3_6 * (16 + t0_2_13);
t0_3_6 += (1 - rt0_3_6) * t0_2_12;

int rt0_3_7 = (input[i] > (239+offset0));
int t0_3_7 = rt0_3_7 * (16 + t0_2_15);
t0_3_7 += (1 - rt0_3_7) * t0_2_14;

int rt0_3_8 = (input[i] > (271+offset0));
int t0_3_8 = rt0_3_8 * (16 + t0_2_17);
t0_3_8 += (1 - rt0_3_8) * t0_2_16;

int rt0_3_9 = (input[i] > (303+offset0));
int t0_3_9 = rt0_3_9 * (16 + t0_2_19);
t0_3_9 += (1 - rt0_3_9) * t0_2_18;

int rt0_3_10 = (input[i] > (335+offset0));
int t0_3_10 = rt0_3_10 * (16 + t0_2_21);
t0_3_10 += (1 - rt0_3_10) * t0_2_20;

int rt0_3_11 = (input[i] > (367+offset0));
int t0_3_11 = rt0_3_11 * (16 + t0_2_23);
t0_3_11 += (1 - rt0_3_11) * t0_2_22;

int rt0_3_12 = (input[i] > (399+offset0));
int t0_3_12 = rt0_3_12 * (16 + t0_2_25);
t0_3_12 += (1 - rt0_3_12) * t0_2_24;

int rt0_3_13 = (input[i] > (431+offset0));
int t0_3_13 = rt0_3_13 * (16 + t0_2_27);
t0_3_13 += (1 - rt0_3_13) * t0_2_26;

int rt0_3_14 = (input[i] > (463+offset0));
int t0_3_14 = rt0_3_14 * (16 + t0_2_29);
t0_3_14 += (1 - rt0_3_14) * t0_2_28;

int rt0_3_15 = (input[i] > (495+offset0));
int t0_3_15 = rt0_3_15 * (16 + t0_2_31);
t0_3_15 += (1 - rt0_3_15) * t0_2_30;

int rt0_4_0 = (input[i] > (31+offset0));
int t0_4_0 = rt0_4_0 * (32 + t0_3_1);
t0_4_0 += (1 - rt0_4_0) * t0_3_0;

int rt0_4_1 = (input[i] > (95+offset0));
int t0_4_1 = rt0_4_1 * (32 + t0_3_3);
t0_4_1 += (1 - rt0_4_1) * t0_3_2;

int rt0_4_2 = (input[i] > (159+offset0));
int t0_4_2 = rt0_4_2 * (32 + t0_3_5);
t0_4_2 += (1 - rt0_4_2) * t0_3_4;

int rt0_4_3 = (input[i] > (223+offset0));
int t0_4_3 = rt0_4_3 * (32 + t0_3_7);
t0_4_3 += (1 - rt0_4_3) * t0_3_6;

int rt0_4_4 = (input[i] > (287+offset0));
int t0_4_4 = rt0_4_4 * (32 + t0_3_9);
t0_4_4 += (1 - rt0_4_4) * t0_3_8;

int rt0_4_5 = (input[i] > (351+offset0));
int t0_4_5 = rt0_4_5 * (32 + t0_3_11);
t0_4_5 += (1 - rt0_4_5) * t0_3_10;

int rt0_4_6 = (input[i] > (415+offset0));
int t0_4_6 = rt0_4_6 * (32 + t0_3_13);
t0_4_6 += (1 - rt0_4_6) * t0_3_12;

int rt0_4_7 = (input[i] > (479+offset0));
int t0_4_7 = rt0_4_7 * (32 + t0_3_15);
t0_4_7 += (1 - rt0_4_7) * t0_3_14;

int rt0_5_0 = (input[i] > (63+offset0));
int t0_5_0 = rt0_5_0 * (64 + t0_4_1);
t0_5_0 += (1 - rt0_5_0) * t0_4_0;

int rt0_5_1 = (input[i] > (191+offset0));
int t0_5_1 = rt0_5_1 * (64 + t0_4_3);
t0_5_1 += (1 - rt0_5_1) * t0_4_2;

int rt0_5_2 = (input[i] > (319+offset0));
int t0_5_2 = rt0_5_2 * (64 + t0_4_5);
t0_5_2 += (1 - rt0_5_2) * t0_4_4;

int rt0_5_3 = (input[i] > (447+offset0));
int t0_5_3 = rt0_5_3 * (64 + t0_4_7);
t0_5_3 += (1 - rt0_5_3) * t0_4_6;

int rt0_6_0 = (input[i] > (127+offset0));
int t0_6_0 = rt0_6_0 * (128 + t0_5_1);
t0_6_0 += (1 - rt0_6_0) * t0_5_0;

int rt0_6_1 = (input[i] > (383+offset0));
int t0_6_1 = rt0_6_1 * (128 + t0_5_3);
t0_6_1 += (1 - rt0_6_1) * t0_5_2;

int rt0_7_0 = (input[i] > (255+offset0));
int t0_7_0 = rt0_7_0 * (256 + t0_6_1);
t0_7_0 += (1 - rt0_7_0) * t0_6_0;

vR[i] = t0_7_0;
	}
}

int main(int argc, char **argv)
{
	float elapsed_time;
	// set up device
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("[CUDA - Random Forest]: EXP %s\n", EXP_NAME);
	printf("[CUDA - Random Forest]: Using Device %d: %s\n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));

	// set up data size of vectors
	int nElem = N_ELEM;
	printf("[CUDA - Random Forest]: Vector Size %d\n", nElem);

	// malloc host memory
	size_t nBytes = nElem * sizeof(float);
	float *h_vR;
	h_vR = (float *)malloc(nBytes);
	float *h_input;
	h_input = (float *)malloc(nBytes);

	printf("[CUDA - Random Forest]: Start Reading Dataset\n");
	initData(h_input, nElem);
	printf("[CUDA - Random Forest]: Finished Reading Dataset\n");
	memset(h_vR, 0, nBytes);

	// malloc device global memory
	float *d_vR;
	CHECK(hipMalloc((float **)&d_vR, nBytes));
	float *d_input;
	CHECK(hipMalloc((float **)&d_input, nBytes));

	// transfer data from host to device
	CHECK(hipMemcpy(d_input, h_input, nBytes, hipMemcpyHostToDevice));

	// invoke kernel at host side
	int iLen = 512;
	dim3 block(iLen);
	dim3 grid((nElem + block.x - 1) / block.x);
	hipEvent_t start, stop;
	CHECK(hipEventCreate(&start));
	CHECK(hipEventCreate(&stop));

	// record start event
	CHECK(hipEventRecord(start, 0));
#if defined(_2TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, d_vR);
#elif defined(_3TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, d_vR);
#elif defined(_4TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, d_vR);
#else
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, d_vR);
#endif
	CHECK(hipEventRecord(stop, 0));
	CHECK(hipEventSynchronize(stop));
	hipDeviceSynchronize();

	// calculate elapsed time
	CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
	printf("[CUDA - Random Forest]: %s - execution time = %.6fms\n", EXP_NAME, elapsed_time);
	registerTime(elapsed_time);

	// copy kernel result back to host side
	CHECK(hipMemcpy(h_vR, d_vR, nBytes, hipMemcpyDeviceToHost));
	CHECK(hipGetLastError());

	//Verify the answers
	char flag = 1;
	for (int i = 0; i < nElem; i++)
	{
		if (i < 32)
		{
			printf("%.3f ", h_vR[i]);
		}
		if (h_input[i] != h_vR[i])
		{
			flag = 0;
			break;
		}
	}
	if (flag == 1)
	{
//		printf("Return correct");
	}
	else
	{
//		printf("Error!!");
	}
	printf("\n ");

	// free device memory
	hipFree(d_input);
	hipFree(d_vR);

	// free host memory
	free(h_input);
	free(h_vR);
	return (0);
}

void initData(float *ip, int size)
{
	for (int i = 0; i < size; i++)
	{
		ip[i] = (float)(0);
	}
	return;
}

void registerTime(float value)
{
	printf("[CUDA - Random Forest]: Time spent: %f.\n", value);
}
