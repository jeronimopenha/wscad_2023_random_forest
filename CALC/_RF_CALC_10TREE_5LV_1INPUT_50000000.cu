#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>

#define N_ELEM 50000000

#define WARP_SIZE 32

#define CHECK(call)                                                \
	{                                                              \
		const hipError_t error = call;                            \
		if (error != hipSuccess)                                  \
		{                                                          \
			fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
			fprintf(stderr, "code: %d, reason: %s\n", error,       \
					hipGetErrorString(error));                    \
		}                                                          \
	}

void initData(float *ip, int size);
void registerTime(float value);

#define _MOD 32
#define _10TREE
#define EXP_NAME "_RF_CALC_10TREE_5LV_1INPUT_50000000"

// RF_with_if

__global__ void RF_kernel(float *input, const int n, const int offset0, const int offset1, const int offset2, const int offset3, const int offset4, const int offset5, const int offset6, const int offset7, const int offset8, const int offset9, float *vR)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
int rt0_0_0 = (input[i] > (1+offset0));
int t0_0_0 = rt0_0_0 * (2 + (input[i] > (2+offset0)));
t0_0_0 += (1 - rt0_0_0) * (input[i] > (0+offset0));

int rt0_0_1 = (input[i] > (5+offset0));
int t0_0_1 = rt0_0_1 * (2 + (input[i] > (6+offset0)));
t0_0_1 += (1 - rt0_0_1) * (input[i] > (4+offset0));

int rt0_0_2 = (input[i] > (9+offset0));
int t0_0_2 = rt0_0_2 * (2 + (input[i] > (10+offset0)));
t0_0_2 += (1 - rt0_0_2) * (input[i] > (8+offset0));

int rt0_0_3 = (input[i] > (13+offset0));
int t0_0_3 = rt0_0_3 * (2 + (input[i] > (14+offset0)));
t0_0_3 += (1 - rt0_0_3) * (input[i] > (12+offset0));

int rt0_0_4 = (input[i] > (17+offset0));
int t0_0_4 = rt0_0_4 * (2 + (input[i] > (18+offset0)));
t0_0_4 += (1 - rt0_0_4) * (input[i] > (16+offset0));

int rt0_0_5 = (input[i] > (21+offset0));
int t0_0_5 = rt0_0_5 * (2 + (input[i] > (22+offset0)));
t0_0_5 += (1 - rt0_0_5) * (input[i] > (20+offset0));

int rt0_0_6 = (input[i] > (25+offset0));
int t0_0_6 = rt0_0_6 * (2 + (input[i] > (26+offset0)));
t0_0_6 += (1 - rt0_0_6) * (input[i] > (24+offset0));

int rt0_0_7 = (input[i] > (29+offset0));
int t0_0_7 = rt0_0_7 * (2 + (input[i] > (30+offset0)));
t0_0_7 += (1 - rt0_0_7) * (input[i] > (28+offset0));

int rt0_1_0 = (input[i] > (3+offset0));
int t0_1_0 = rt0_1_0 * (4 + t0_0_1);
t0_1_0 += (1 - rt0_1_0) * t0_0_0;

int rt0_1_1 = (input[i] > (11+offset0));
int t0_1_1 = rt0_1_1 * (4 + t0_0_3);
t0_1_1 += (1 - rt0_1_1) * t0_0_2;

int rt0_1_2 = (input[i] > (19+offset0));
int t0_1_2 = rt0_1_2 * (4 + t0_0_5);
t0_1_2 += (1 - rt0_1_2) * t0_0_4;

int rt0_1_3 = (input[i] > (27+offset0));
int t0_1_3 = rt0_1_3 * (4 + t0_0_7);
t0_1_3 += (1 - rt0_1_3) * t0_0_6;

int rt0_2_0 = (input[i] > (7+offset0));
int t0_2_0 = rt0_2_0 * (8 + t0_1_1);
t0_2_0 += (1 - rt0_2_0) * t0_1_0;

int rt0_2_1 = (input[i] > (23+offset0));
int t0_2_1 = rt0_2_1 * (8 + t0_1_3);
t0_2_1 += (1 - rt0_2_1) * t0_1_2;

int rt0_3_0 = (input[i] > (15+offset0));
int t0_3_0 = rt0_3_0 * (16 + t0_2_1);
t0_3_0 += (1 - rt0_3_0) * t0_2_0;

int rt1_0_0 = (input[i] > (1+offset1));
int t1_0_0 = rt1_0_0 * (2 + (input[i] > (2+offset1)));
t1_0_0 += (1 - rt1_0_0) * (input[i] > (0+offset1));

int rt1_0_1 = (input[i] > (5+offset1));
int t1_0_1 = rt1_0_1 * (2 + (input[i] > (6+offset1)));
t1_0_1 += (1 - rt1_0_1) * (input[i] > (4+offset1));

int rt1_0_2 = (input[i] > (9+offset1));
int t1_0_2 = rt1_0_2 * (2 + (input[i] > (10+offset1)));
t1_0_2 += (1 - rt1_0_2) * (input[i] > (8+offset1));

int rt1_0_3 = (input[i] > (13+offset1));
int t1_0_3 = rt1_0_3 * (2 + (input[i] > (14+offset1)));
t1_0_3 += (1 - rt1_0_3) * (input[i] > (12+offset1));

int rt1_0_4 = (input[i] > (17+offset1));
int t1_0_4 = rt1_0_4 * (2 + (input[i] > (18+offset1)));
t1_0_4 += (1 - rt1_0_4) * (input[i] > (16+offset1));

int rt1_0_5 = (input[i] > (21+offset1));
int t1_0_5 = rt1_0_5 * (2 + (input[i] > (22+offset1)));
t1_0_5 += (1 - rt1_0_5) * (input[i] > (20+offset1));

int rt1_0_6 = (input[i] > (25+offset1));
int t1_0_6 = rt1_0_6 * (2 + (input[i] > (26+offset1)));
t1_0_6 += (1 - rt1_0_6) * (input[i] > (24+offset1));

int rt1_0_7 = (input[i] > (29+offset1));
int t1_0_7 = rt1_0_7 * (2 + (input[i] > (30+offset1)));
t1_0_7 += (1 - rt1_0_7) * (input[i] > (28+offset1));

int rt1_1_0 = (input[i] > (3+offset1));
int t1_1_0 = rt1_1_0 * (4 + t1_0_1);
t1_1_0 += (1 - rt1_1_0) * t1_0_0;

int rt1_1_1 = (input[i] > (11+offset1));
int t1_1_1 = rt1_1_1 * (4 + t1_0_3);
t1_1_1 += (1 - rt1_1_1) * t1_0_2;

int rt1_1_2 = (input[i] > (19+offset1));
int t1_1_2 = rt1_1_2 * (4 + t1_0_5);
t1_1_2 += (1 - rt1_1_2) * t1_0_4;

int rt1_1_3 = (input[i] > (27+offset1));
int t1_1_3 = rt1_1_3 * (4 + t1_0_7);
t1_1_3 += (1 - rt1_1_3) * t1_0_6;

int rt1_2_0 = (input[i] > (7+offset1));
int t1_2_0 = rt1_2_0 * (8 + t1_1_1);
t1_2_0 += (1 - rt1_2_0) * t1_1_0;

int rt1_2_1 = (input[i] > (23+offset1));
int t1_2_1 = rt1_2_1 * (8 + t1_1_3);
t1_2_1 += (1 - rt1_2_1) * t1_1_2;

int rt1_3_0 = (input[i] > (15+offset1));
int t1_3_0 = rt1_3_0 * (16 + t1_2_1);
t1_3_0 += (1 - rt1_3_0) * t1_2_0;

int rt2_0_0 = (input[i] > (1+offset2));
int t2_0_0 = rt2_0_0 * (2 + (input[i] > (2+offset2)));
t2_0_0 += (1 - rt2_0_0) * (input[i] > (0+offset2));

int rt2_0_1 = (input[i] > (5+offset2));
int t2_0_1 = rt2_0_1 * (2 + (input[i] > (6+offset2)));
t2_0_1 += (1 - rt2_0_1) * (input[i] > (4+offset2));

int rt2_0_2 = (input[i] > (9+offset2));
int t2_0_2 = rt2_0_2 * (2 + (input[i] > (10+offset2)));
t2_0_2 += (1 - rt2_0_2) * (input[i] > (8+offset2));

int rt2_0_3 = (input[i] > (13+offset2));
int t2_0_3 = rt2_0_3 * (2 + (input[i] > (14+offset2)));
t2_0_3 += (1 - rt2_0_3) * (input[i] > (12+offset2));

int rt2_0_4 = (input[i] > (17+offset2));
int t2_0_4 = rt2_0_4 * (2 + (input[i] > (18+offset2)));
t2_0_4 += (1 - rt2_0_4) * (input[i] > (16+offset2));

int rt2_0_5 = (input[i] > (21+offset2));
int t2_0_5 = rt2_0_5 * (2 + (input[i] > (22+offset2)));
t2_0_5 += (1 - rt2_0_5) * (input[i] > (20+offset2));

int rt2_0_6 = (input[i] > (25+offset2));
int t2_0_6 = rt2_0_6 * (2 + (input[i] > (26+offset2)));
t2_0_6 += (1 - rt2_0_6) * (input[i] > (24+offset2));

int rt2_0_7 = (input[i] > (29+offset2));
int t2_0_7 = rt2_0_7 * (2 + (input[i] > (30+offset2)));
t2_0_7 += (1 - rt2_0_7) * (input[i] > (28+offset2));

int rt2_1_0 = (input[i] > (3+offset2));
int t2_1_0 = rt2_1_0 * (4 + t2_0_1);
t2_1_0 += (1 - rt2_1_0) * t2_0_0;

int rt2_1_1 = (input[i] > (11+offset2));
int t2_1_1 = rt2_1_1 * (4 + t2_0_3);
t2_1_1 += (1 - rt2_1_1) * t2_0_2;

int rt2_1_2 = (input[i] > (19+offset2));
int t2_1_2 = rt2_1_2 * (4 + t2_0_5);
t2_1_2 += (1 - rt2_1_2) * t2_0_4;

int rt2_1_3 = (input[i] > (27+offset2));
int t2_1_3 = rt2_1_3 * (4 + t2_0_7);
t2_1_3 += (1 - rt2_1_3) * t2_0_6;

int rt2_2_0 = (input[i] > (7+offset2));
int t2_2_0 = rt2_2_0 * (8 + t2_1_1);
t2_2_0 += (1 - rt2_2_0) * t2_1_0;

int rt2_2_1 = (input[i] > (23+offset2));
int t2_2_1 = rt2_2_1 * (8 + t2_1_3);
t2_2_1 += (1 - rt2_2_1) * t2_1_2;

int rt2_3_0 = (input[i] > (15+offset2));
int t2_3_0 = rt2_3_0 * (16 + t2_2_1);
t2_3_0 += (1 - rt2_3_0) * t2_2_0;

int rt3_0_0 = (input[i] > (1+offset3));
int t3_0_0 = rt3_0_0 * (2 + (input[i] > (2+offset3)));
t3_0_0 += (1 - rt3_0_0) * (input[i] > (0+offset3));

int rt3_0_1 = (input[i] > (5+offset3));
int t3_0_1 = rt3_0_1 * (2 + (input[i] > (6+offset3)));
t3_0_1 += (1 - rt3_0_1) * (input[i] > (4+offset3));

int rt3_0_2 = (input[i] > (9+offset3));
int t3_0_2 = rt3_0_2 * (2 + (input[i] > (10+offset3)));
t3_0_2 += (1 - rt3_0_2) * (input[i] > (8+offset3));

int rt3_0_3 = (input[i] > (13+offset3));
int t3_0_3 = rt3_0_3 * (2 + (input[i] > (14+offset3)));
t3_0_3 += (1 - rt3_0_3) * (input[i] > (12+offset3));

int rt3_0_4 = (input[i] > (17+offset3));
int t3_0_4 = rt3_0_4 * (2 + (input[i] > (18+offset3)));
t3_0_4 += (1 - rt3_0_4) * (input[i] > (16+offset3));

int rt3_0_5 = (input[i] > (21+offset3));
int t3_0_5 = rt3_0_5 * (2 + (input[i] > (22+offset3)));
t3_0_5 += (1 - rt3_0_5) * (input[i] > (20+offset3));

int rt3_0_6 = (input[i] > (25+offset3));
int t3_0_6 = rt3_0_6 * (2 + (input[i] > (26+offset3)));
t3_0_6 += (1 - rt3_0_6) * (input[i] > (24+offset3));

int rt3_0_7 = (input[i] > (29+offset3));
int t3_0_7 = rt3_0_7 * (2 + (input[i] > (30+offset3)));
t3_0_7 += (1 - rt3_0_7) * (input[i] > (28+offset3));

int rt3_1_0 = (input[i] > (3+offset3));
int t3_1_0 = rt3_1_0 * (4 + t3_0_1);
t3_1_0 += (1 - rt3_1_0) * t3_0_0;

int rt3_1_1 = (input[i] > (11+offset3));
int t3_1_1 = rt3_1_1 * (4 + t3_0_3);
t3_1_1 += (1 - rt3_1_1) * t3_0_2;

int rt3_1_2 = (input[i] > (19+offset3));
int t3_1_2 = rt3_1_2 * (4 + t3_0_5);
t3_1_2 += (1 - rt3_1_2) * t3_0_4;

int rt3_1_3 = (input[i] > (27+offset3));
int t3_1_3 = rt3_1_3 * (4 + t3_0_7);
t3_1_3 += (1 - rt3_1_3) * t3_0_6;

int rt3_2_0 = (input[i] > (7+offset3));
int t3_2_0 = rt3_2_0 * (8 + t3_1_1);
t3_2_0 += (1 - rt3_2_0) * t3_1_0;

int rt3_2_1 = (input[i] > (23+offset3));
int t3_2_1 = rt3_2_1 * (8 + t3_1_3);
t3_2_1 += (1 - rt3_2_1) * t3_1_2;

int rt3_3_0 = (input[i] > (15+offset3));
int t3_3_0 = rt3_3_0 * (16 + t3_2_1);
t3_3_0 += (1 - rt3_3_0) * t3_2_0;

int rt4_0_0 = (input[i] > (1+offset4));
int t4_0_0 = rt4_0_0 * (2 + (input[i] > (2+offset4)));
t4_0_0 += (1 - rt4_0_0) * (input[i] > (0+offset4));

int rt4_0_1 = (input[i] > (5+offset4));
int t4_0_1 = rt4_0_1 * (2 + (input[i] > (6+offset4)));
t4_0_1 += (1 - rt4_0_1) * (input[i] > (4+offset4));

int rt4_0_2 = (input[i] > (9+offset4));
int t4_0_2 = rt4_0_2 * (2 + (input[i] > (10+offset4)));
t4_0_2 += (1 - rt4_0_2) * (input[i] > (8+offset4));

int rt4_0_3 = (input[i] > (13+offset4));
int t4_0_3 = rt4_0_3 * (2 + (input[i] > (14+offset4)));
t4_0_3 += (1 - rt4_0_3) * (input[i] > (12+offset4));

int rt4_0_4 = (input[i] > (17+offset4));
int t4_0_4 = rt4_0_4 * (2 + (input[i] > (18+offset4)));
t4_0_4 += (1 - rt4_0_4) * (input[i] > (16+offset4));

int rt4_0_5 = (input[i] > (21+offset4));
int t4_0_5 = rt4_0_5 * (2 + (input[i] > (22+offset4)));
t4_0_5 += (1 - rt4_0_5) * (input[i] > (20+offset4));

int rt4_0_6 = (input[i] > (25+offset4));
int t4_0_6 = rt4_0_6 * (2 + (input[i] > (26+offset4)));
t4_0_6 += (1 - rt4_0_6) * (input[i] > (24+offset4));

int rt4_0_7 = (input[i] > (29+offset4));
int t4_0_7 = rt4_0_7 * (2 + (input[i] > (30+offset4)));
t4_0_7 += (1 - rt4_0_7) * (input[i] > (28+offset4));

int rt4_1_0 = (input[i] > (3+offset4));
int t4_1_0 = rt4_1_0 * (4 + t4_0_1);
t4_1_0 += (1 - rt4_1_0) * t4_0_0;

int rt4_1_1 = (input[i] > (11+offset4));
int t4_1_1 = rt4_1_1 * (4 + t4_0_3);
t4_1_1 += (1 - rt4_1_1) * t4_0_2;

int rt4_1_2 = (input[i] > (19+offset4));
int t4_1_2 = rt4_1_2 * (4 + t4_0_5);
t4_1_2 += (1 - rt4_1_2) * t4_0_4;

int rt4_1_3 = (input[i] > (27+offset4));
int t4_1_3 = rt4_1_3 * (4 + t4_0_7);
t4_1_3 += (1 - rt4_1_3) * t4_0_6;

int rt4_2_0 = (input[i] > (7+offset4));
int t4_2_0 = rt4_2_0 * (8 + t4_1_1);
t4_2_0 += (1 - rt4_2_0) * t4_1_0;

int rt4_2_1 = (input[i] > (23+offset4));
int t4_2_1 = rt4_2_1 * (8 + t4_1_3);
t4_2_1 += (1 - rt4_2_1) * t4_1_2;

int rt4_3_0 = (input[i] > (15+offset4));
int t4_3_0 = rt4_3_0 * (16 + t4_2_1);
t4_3_0 += (1 - rt4_3_0) * t4_2_0;

int rt5_0_0 = (input[i] > (1+offset5));
int t5_0_0 = rt5_0_0 * (2 + (input[i] > (2+offset5)));
t5_0_0 += (1 - rt5_0_0) * (input[i] > (0+offset5));

int rt5_0_1 = (input[i] > (5+offset5));
int t5_0_1 = rt5_0_1 * (2 + (input[i] > (6+offset5)));
t5_0_1 += (1 - rt5_0_1) * (input[i] > (4+offset5));

int rt5_0_2 = (input[i] > (9+offset5));
int t5_0_2 = rt5_0_2 * (2 + (input[i] > (10+offset5)));
t5_0_2 += (1 - rt5_0_2) * (input[i] > (8+offset5));

int rt5_0_3 = (input[i] > (13+offset5));
int t5_0_3 = rt5_0_3 * (2 + (input[i] > (14+offset5)));
t5_0_3 += (1 - rt5_0_3) * (input[i] > (12+offset5));

int rt5_0_4 = (input[i] > (17+offset5));
int t5_0_4 = rt5_0_4 * (2 + (input[i] > (18+offset5)));
t5_0_4 += (1 - rt5_0_4) * (input[i] > (16+offset5));

int rt5_0_5 = (input[i] > (21+offset5));
int t5_0_5 = rt5_0_5 * (2 + (input[i] > (22+offset5)));
t5_0_5 += (1 - rt5_0_5) * (input[i] > (20+offset5));

int rt5_0_6 = (input[i] > (25+offset5));
int t5_0_6 = rt5_0_6 * (2 + (input[i] > (26+offset5)));
t5_0_6 += (1 - rt5_0_6) * (input[i] > (24+offset5));

int rt5_0_7 = (input[i] > (29+offset5));
int t5_0_7 = rt5_0_7 * (2 + (input[i] > (30+offset5)));
t5_0_7 += (1 - rt5_0_7) * (input[i] > (28+offset5));

int rt5_1_0 = (input[i] > (3+offset5));
int t5_1_0 = rt5_1_0 * (4 + t5_0_1);
t5_1_0 += (1 - rt5_1_0) * t5_0_0;

int rt5_1_1 = (input[i] > (11+offset5));
int t5_1_1 = rt5_1_1 * (4 + t5_0_3);
t5_1_1 += (1 - rt5_1_1) * t5_0_2;

int rt5_1_2 = (input[i] > (19+offset5));
int t5_1_2 = rt5_1_2 * (4 + t5_0_5);
t5_1_2 += (1 - rt5_1_2) * t5_0_4;

int rt5_1_3 = (input[i] > (27+offset5));
int t5_1_3 = rt5_1_3 * (4 + t5_0_7);
t5_1_3 += (1 - rt5_1_3) * t5_0_6;

int rt5_2_0 = (input[i] > (7+offset5));
int t5_2_0 = rt5_2_0 * (8 + t5_1_1);
t5_2_0 += (1 - rt5_2_0) * t5_1_0;

int rt5_2_1 = (input[i] > (23+offset5));
int t5_2_1 = rt5_2_1 * (8 + t5_1_3);
t5_2_1 += (1 - rt5_2_1) * t5_1_2;

int rt5_3_0 = (input[i] > (15+offset5));
int t5_3_0 = rt5_3_0 * (16 + t5_2_1);
t5_3_0 += (1 - rt5_3_0) * t5_2_0;

int rt6_0_0 = (input[i] > (1+offset6));
int t6_0_0 = rt6_0_0 * (2 + (input[i] > (2+offset6)));
t6_0_0 += (1 - rt6_0_0) * (input[i] > (0+offset6));

int rt6_0_1 = (input[i] > (5+offset6));
int t6_0_1 = rt6_0_1 * (2 + (input[i] > (6+offset6)));
t6_0_1 += (1 - rt6_0_1) * (input[i] > (4+offset6));

int rt6_0_2 = (input[i] > (9+offset6));
int t6_0_2 = rt6_0_2 * (2 + (input[i] > (10+offset6)));
t6_0_2 += (1 - rt6_0_2) * (input[i] > (8+offset6));

int rt6_0_3 = (input[i] > (13+offset6));
int t6_0_3 = rt6_0_3 * (2 + (input[i] > (14+offset6)));
t6_0_3 += (1 - rt6_0_3) * (input[i] > (12+offset6));

int rt6_0_4 = (input[i] > (17+offset6));
int t6_0_4 = rt6_0_4 * (2 + (input[i] > (18+offset6)));
t6_0_4 += (1 - rt6_0_4) * (input[i] > (16+offset6));

int rt6_0_5 = (input[i] > (21+offset6));
int t6_0_5 = rt6_0_5 * (2 + (input[i] > (22+offset6)));
t6_0_5 += (1 - rt6_0_5) * (input[i] > (20+offset6));

int rt6_0_6 = (input[i] > (25+offset6));
int t6_0_6 = rt6_0_6 * (2 + (input[i] > (26+offset6)));
t6_0_6 += (1 - rt6_0_6) * (input[i] > (24+offset6));

int rt6_0_7 = (input[i] > (29+offset6));
int t6_0_7 = rt6_0_7 * (2 + (input[i] > (30+offset6)));
t6_0_7 += (1 - rt6_0_7) * (input[i] > (28+offset6));

int rt6_1_0 = (input[i] > (3+offset6));
int t6_1_0 = rt6_1_0 * (4 + t6_0_1);
t6_1_0 += (1 - rt6_1_0) * t6_0_0;

int rt6_1_1 = (input[i] > (11+offset6));
int t6_1_1 = rt6_1_1 * (4 + t6_0_3);
t6_1_1 += (1 - rt6_1_1) * t6_0_2;

int rt6_1_2 = (input[i] > (19+offset6));
int t6_1_2 = rt6_1_2 * (4 + t6_0_5);
t6_1_2 += (1 - rt6_1_2) * t6_0_4;

int rt6_1_3 = (input[i] > (27+offset6));
int t6_1_3 = rt6_1_3 * (4 + t6_0_7);
t6_1_3 += (1 - rt6_1_3) * t6_0_6;

int rt6_2_0 = (input[i] > (7+offset6));
int t6_2_0 = rt6_2_0 * (8 + t6_1_1);
t6_2_0 += (1 - rt6_2_0) * t6_1_0;

int rt6_2_1 = (input[i] > (23+offset6));
int t6_2_1 = rt6_2_1 * (8 + t6_1_3);
t6_2_1 += (1 - rt6_2_1) * t6_1_2;

int rt6_3_0 = (input[i] > (15+offset6));
int t6_3_0 = rt6_3_0 * (16 + t6_2_1);
t6_3_0 += (1 - rt6_3_0) * t6_2_0;

int rt7_0_0 = (input[i] > (1+offset7));
int t7_0_0 = rt7_0_0 * (2 + (input[i] > (2+offset7)));
t7_0_0 += (1 - rt7_0_0) * (input[i] > (0+offset7));

int rt7_0_1 = (input[i] > (5+offset7));
int t7_0_1 = rt7_0_1 * (2 + (input[i] > (6+offset7)));
t7_0_1 += (1 - rt7_0_1) * (input[i] > (4+offset7));

int rt7_0_2 = (input[i] > (9+offset7));
int t7_0_2 = rt7_0_2 * (2 + (input[i] > (10+offset7)));
t7_0_2 += (1 - rt7_0_2) * (input[i] > (8+offset7));

int rt7_0_3 = (input[i] > (13+offset7));
int t7_0_3 = rt7_0_3 * (2 + (input[i] > (14+offset7)));
t7_0_3 += (1 - rt7_0_3) * (input[i] > (12+offset7));

int rt7_0_4 = (input[i] > (17+offset7));
int t7_0_4 = rt7_0_4 * (2 + (input[i] > (18+offset7)));
t7_0_4 += (1 - rt7_0_4) * (input[i] > (16+offset7));

int rt7_0_5 = (input[i] > (21+offset7));
int t7_0_5 = rt7_0_5 * (2 + (input[i] > (22+offset7)));
t7_0_5 += (1 - rt7_0_5) * (input[i] > (20+offset7));

int rt7_0_6 = (input[i] > (25+offset7));
int t7_0_6 = rt7_0_6 * (2 + (input[i] > (26+offset7)));
t7_0_6 += (1 - rt7_0_6) * (input[i] > (24+offset7));

int rt7_0_7 = (input[i] > (29+offset7));
int t7_0_7 = rt7_0_7 * (2 + (input[i] > (30+offset7)));
t7_0_7 += (1 - rt7_0_7) * (input[i] > (28+offset7));

int rt7_1_0 = (input[i] > (3+offset7));
int t7_1_0 = rt7_1_0 * (4 + t7_0_1);
t7_1_0 += (1 - rt7_1_0) * t7_0_0;

int rt7_1_1 = (input[i] > (11+offset7));
int t7_1_1 = rt7_1_1 * (4 + t7_0_3);
t7_1_1 += (1 - rt7_1_1) * t7_0_2;

int rt7_1_2 = (input[i] > (19+offset7));
int t7_1_2 = rt7_1_2 * (4 + t7_0_5);
t7_1_2 += (1 - rt7_1_2) * t7_0_4;

int rt7_1_3 = (input[i] > (27+offset7));
int t7_1_3 = rt7_1_3 * (4 + t7_0_7);
t7_1_3 += (1 - rt7_1_3) * t7_0_6;

int rt7_2_0 = (input[i] > (7+offset7));
int t7_2_0 = rt7_2_0 * (8 + t7_1_1);
t7_2_0 += (1 - rt7_2_0) * t7_1_0;

int rt7_2_1 = (input[i] > (23+offset7));
int t7_2_1 = rt7_2_1 * (8 + t7_1_3);
t7_2_1 += (1 - rt7_2_1) * t7_1_2;

int rt7_3_0 = (input[i] > (15+offset7));
int t7_3_0 = rt7_3_0 * (16 + t7_2_1);
t7_3_0 += (1 - rt7_3_0) * t7_2_0;

int rt8_0_0 = (input[i] > (1+offset8));
int t8_0_0 = rt8_0_0 * (2 + (input[i] > (2+offset8)));
t8_0_0 += (1 - rt8_0_0) * (input[i] > (0+offset8));

int rt8_0_1 = (input[i] > (5+offset8));
int t8_0_1 = rt8_0_1 * (2 + (input[i] > (6+offset8)));
t8_0_1 += (1 - rt8_0_1) * (input[i] > (4+offset8));

int rt8_0_2 = (input[i] > (9+offset8));
int t8_0_2 = rt8_0_2 * (2 + (input[i] > (10+offset8)));
t8_0_2 += (1 - rt8_0_2) * (input[i] > (8+offset8));

int rt8_0_3 = (input[i] > (13+offset8));
int t8_0_3 = rt8_0_3 * (2 + (input[i] > (14+offset8)));
t8_0_3 += (1 - rt8_0_3) * (input[i] > (12+offset8));

int rt8_0_4 = (input[i] > (17+offset8));
int t8_0_4 = rt8_0_4 * (2 + (input[i] > (18+offset8)));
t8_0_4 += (1 - rt8_0_4) * (input[i] > (16+offset8));

int rt8_0_5 = (input[i] > (21+offset8));
int t8_0_5 = rt8_0_5 * (2 + (input[i] > (22+offset8)));
t8_0_5 += (1 - rt8_0_5) * (input[i] > (20+offset8));

int rt8_0_6 = (input[i] > (25+offset8));
int t8_0_6 = rt8_0_6 * (2 + (input[i] > (26+offset8)));
t8_0_6 += (1 - rt8_0_6) * (input[i] > (24+offset8));

int rt8_0_7 = (input[i] > (29+offset8));
int t8_0_7 = rt8_0_7 * (2 + (input[i] > (30+offset8)));
t8_0_7 += (1 - rt8_0_7) * (input[i] > (28+offset8));

int rt8_1_0 = (input[i] > (3+offset8));
int t8_1_0 = rt8_1_0 * (4 + t8_0_1);
t8_1_0 += (1 - rt8_1_0) * t8_0_0;

int rt8_1_1 = (input[i] > (11+offset8));
int t8_1_1 = rt8_1_1 * (4 + t8_0_3);
t8_1_1 += (1 - rt8_1_1) * t8_0_2;

int rt8_1_2 = (input[i] > (19+offset8));
int t8_1_2 = rt8_1_2 * (4 + t8_0_5);
t8_1_2 += (1 - rt8_1_2) * t8_0_4;

int rt8_1_3 = (input[i] > (27+offset8));
int t8_1_3 = rt8_1_3 * (4 + t8_0_7);
t8_1_3 += (1 - rt8_1_3) * t8_0_6;

int rt8_2_0 = (input[i] > (7+offset8));
int t8_2_0 = rt8_2_0 * (8 + t8_1_1);
t8_2_0 += (1 - rt8_2_0) * t8_1_0;

int rt8_2_1 = (input[i] > (23+offset8));
int t8_2_1 = rt8_2_1 * (8 + t8_1_3);
t8_2_1 += (1 - rt8_2_1) * t8_1_2;

int rt8_3_0 = (input[i] > (15+offset8));
int t8_3_0 = rt8_3_0 * (16 + t8_2_1);
t8_3_0 += (1 - rt8_3_0) * t8_2_0;

int rt9_0_0 = (input[i] > (1+offset9));
int t9_0_0 = rt9_0_0 * (2 + (input[i] > (2+offset9)));
t9_0_0 += (1 - rt9_0_0) * (input[i] > (0+offset9));

int rt9_0_1 = (input[i] > (5+offset9));
int t9_0_1 = rt9_0_1 * (2 + (input[i] > (6+offset9)));
t9_0_1 += (1 - rt9_0_1) * (input[i] > (4+offset9));

int rt9_0_2 = (input[i] > (9+offset9));
int t9_0_2 = rt9_0_2 * (2 + (input[i] > (10+offset9)));
t9_0_2 += (1 - rt9_0_2) * (input[i] > (8+offset9));

int rt9_0_3 = (input[i] > (13+offset9));
int t9_0_3 = rt9_0_3 * (2 + (input[i] > (14+offset9)));
t9_0_3 += (1 - rt9_0_3) * (input[i] > (12+offset9));

int rt9_0_4 = (input[i] > (17+offset9));
int t9_0_4 = rt9_0_4 * (2 + (input[i] > (18+offset9)));
t9_0_4 += (1 - rt9_0_4) * (input[i] > (16+offset9));

int rt9_0_5 = (input[i] > (21+offset9));
int t9_0_5 = rt9_0_5 * (2 + (input[i] > (22+offset9)));
t9_0_5 += (1 - rt9_0_5) * (input[i] > (20+offset9));

int rt9_0_6 = (input[i] > (25+offset9));
int t9_0_6 = rt9_0_6 * (2 + (input[i] > (26+offset9)));
t9_0_6 += (1 - rt9_0_6) * (input[i] > (24+offset9));

int rt9_0_7 = (input[i] > (29+offset9));
int t9_0_7 = rt9_0_7 * (2 + (input[i] > (30+offset9)));
t9_0_7 += (1 - rt9_0_7) * (input[i] > (28+offset9));

int rt9_1_0 = (input[i] > (3+offset9));
int t9_1_0 = rt9_1_0 * (4 + t9_0_1);
t9_1_0 += (1 - rt9_1_0) * t9_0_0;

int rt9_1_1 = (input[i] > (11+offset9));
int t9_1_1 = rt9_1_1 * (4 + t9_0_3);
t9_1_1 += (1 - rt9_1_1) * t9_0_2;

int rt9_1_2 = (input[i] > (19+offset9));
int t9_1_2 = rt9_1_2 * (4 + t9_0_5);
t9_1_2 += (1 - rt9_1_2) * t9_0_4;

int rt9_1_3 = (input[i] > (27+offset9));
int t9_1_3 = rt9_1_3 * (4 + t9_0_7);
t9_1_3 += (1 - rt9_1_3) * t9_0_6;

int rt9_2_0 = (input[i] > (7+offset9));
int t9_2_0 = rt9_2_0 * (8 + t9_1_1);
t9_2_0 += (1 - rt9_2_0) * t9_1_0;

int rt9_2_1 = (input[i] > (23+offset9));
int t9_2_1 = rt9_2_1 * (8 + t9_1_3);
t9_2_1 += (1 - rt9_2_1) * t9_1_2;

int rt9_3_0 = (input[i] > (15+offset9));
int t9_3_0 = rt9_3_0 * (16 + t9_2_1);
t9_3_0 += (1 - rt9_3_0) * t9_2_0;

vR[i] = t0_3_0+ (t0_3_0 == t1_3_0)+ (t0_3_0 == t2_3_0)+ (t0_3_0 == t3_3_0)+ (t0_3_0 == t4_3_0)+ (t0_3_0 == t5_3_0)+ (t0_3_0 == t6_3_0)+ (t0_3_0 == t7_3_0)+ (t0_3_0 == t8_3_0)+ (t0_3_0 == t9_3_0);
	}
}

int main(int argc, char **argv)
{
	float elapsed_time;
	// set up device
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("[CUDA - Random Forest]: EXP %s\n", EXP_NAME);
	printf("[CUDA - Random Forest]: Using Device %d: %s\n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));

	// set up data size of vectors
	int nElem = N_ELEM;
	printf("[CUDA - Random Forest]: Vector Size %d\n", nElem);

	// malloc host memory
	size_t nBytes = nElem * sizeof(float);
	float *h_vR;
	h_vR = (float *)malloc(nBytes);
	float *h_input;
	h_input = (float *)malloc(nBytes);

	printf("[CUDA - Random Forest]: Start Reading Dataset\n");
	initData(h_input, nElem);
	printf("[CUDA - Random Forest]: Finished Reading Dataset\n");
	memset(h_vR, 0, nBytes);

	// malloc device global memory
	float *d_vR;
	CHECK(hipMalloc((float **)&d_vR, nBytes));
	float *d_input;
	CHECK(hipMalloc((float **)&d_input, nBytes));

	// transfer data from host to device
	CHECK(hipMemcpy(d_input, h_input, nBytes, hipMemcpyHostToDevice));

	// invoke kernel at host side
	int iLen = 512;
	dim3 block(iLen);
	dim3 grid((nElem + block.x - 1) / block.x);
	hipEvent_t start, stop;
	CHECK(hipEventCreate(&start));
	CHECK(hipEventCreate(&stop));

	// record start event
	CHECK(hipEventRecord(start, 0));
#if defined(_2TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, d_vR);
#elif defined(_3TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, d_vR);
#elif defined(_4TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, d_vR);
#elif defined(_5TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, 0, d_vR);
#elif defined(_6TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, 0, 0, d_vR);
#elif defined(_7TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, 0, 0, 0, d_vR);
#elif defined(_8TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, 0, 0, 0, 0, d_vR);
#elif defined(_9TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, 0, 0, 0, 0, 0, d_vR);
#elif defined(_10TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, d_vR);
#else
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, d_vR);
#endif
	CHECK(hipEventRecord(stop, 0));
	CHECK(hipEventSynchronize(stop));
	hipDeviceSynchronize();

	// calculate elapsed time
	CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
	printf("[CUDA - Random Forest]: %s - execution time = %.6fms\n", EXP_NAME, elapsed_time);
	registerTime(elapsed_time);

	// copy kernel result back to host side
	CHECK(hipMemcpy(h_vR, d_vR, nBytes, hipMemcpyDeviceToHost));
	CHECK(hipGetLastError());

	//Verify the answers
	char flag = 1;
	for (int i = 0; i < nElem; i++)
	{
		if (i < 32)
		{
			printf("%.3f ", h_vR[i]);
		}
		if (h_input[i] != h_vR[i])
		{
			flag = 0;
			break;
		}
	}
	if (flag == 1)
	{
//		printf("Return correct");
	}
	else
	{
//		printf("Error!!");
	}
	printf("\n ");

	// free device memory
	hipFree(d_input);
	hipFree(d_vR);

	// free host memory
	free(h_input);
	free(h_vR);
	return (0);
}

void initData(float *ip, int size)
{
	for (int i = 0; i < size; i++)
	{
		ip[i] = (float)(0);
	}
	return;
}

void registerTime(float value)
{
	printf("[CUDA - Random Forest]: Time spent: %f.\n", value);
}
