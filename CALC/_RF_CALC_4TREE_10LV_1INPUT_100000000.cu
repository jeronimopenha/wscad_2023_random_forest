#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>

#define N_ELEM 100000000

#define WARP_SIZE 32

#define CHECK(call)                                                \
	{                                                              \
		const hipError_t error = call;                            \
		if (error != hipSuccess)                                  \
		{                                                          \
			fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
			fprintf(stderr, "code: %d, reason: %s\n", error,       \
					hipGetErrorString(error));                    \
		}                                                          \
	}

void initData(float *ip, int size);
void registerTime(float value);

#define _MOD 1024
#define _4TREE
#define EXP_NAME "_RF_CALC_4TREE_10LV_1INPUT_100000000"

// RF_with_if

__global__ void RF_kernel(float *input, const int n, const int offset0, const int offset1, const int offset2, const int offset3, float *vR)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
int rt0_0_0 = (input[i] > (1+offset0));
int t0_0_0 = rt0_0_0 * (2 + (input[i] > (2+offset0)));
t0_0_0 += (1 - rt0_0_0) * (input[i] > (0+offset0));

int rt0_0_1 = (input[i] > (5+offset0));
int t0_0_1 = rt0_0_1 * (2 + (input[i] > (6+offset0)));
t0_0_1 += (1 - rt0_0_1) * (input[i] > (4+offset0));

int rt0_0_2 = (input[i] > (9+offset0));
int t0_0_2 = rt0_0_2 * (2 + (input[i] > (10+offset0)));
t0_0_2 += (1 - rt0_0_2) * (input[i] > (8+offset0));

int rt0_0_3 = (input[i] > (13+offset0));
int t0_0_3 = rt0_0_3 * (2 + (input[i] > (14+offset0)));
t0_0_3 += (1 - rt0_0_3) * (input[i] > (12+offset0));

int rt0_0_4 = (input[i] > (17+offset0));
int t0_0_4 = rt0_0_4 * (2 + (input[i] > (18+offset0)));
t0_0_4 += (1 - rt0_0_4) * (input[i] > (16+offset0));

int rt0_0_5 = (input[i] > (21+offset0));
int t0_0_5 = rt0_0_5 * (2 + (input[i] > (22+offset0)));
t0_0_5 += (1 - rt0_0_5) * (input[i] > (20+offset0));

int rt0_0_6 = (input[i] > (25+offset0));
int t0_0_6 = rt0_0_6 * (2 + (input[i] > (26+offset0)));
t0_0_6 += (1 - rt0_0_6) * (input[i] > (24+offset0));

int rt0_0_7 = (input[i] > (29+offset0));
int t0_0_7 = rt0_0_7 * (2 + (input[i] > (30+offset0)));
t0_0_7 += (1 - rt0_0_7) * (input[i] > (28+offset0));

int rt0_0_8 = (input[i] > (33+offset0));
int t0_0_8 = rt0_0_8 * (2 + (input[i] > (34+offset0)));
t0_0_8 += (1 - rt0_0_8) * (input[i] > (32+offset0));

int rt0_0_9 = (input[i] > (37+offset0));
int t0_0_9 = rt0_0_9 * (2 + (input[i] > (38+offset0)));
t0_0_9 += (1 - rt0_0_9) * (input[i] > (36+offset0));

int rt0_0_10 = (input[i] > (41+offset0));
int t0_0_10 = rt0_0_10 * (2 + (input[i] > (42+offset0)));
t0_0_10 += (1 - rt0_0_10) * (input[i] > (40+offset0));

int rt0_0_11 = (input[i] > (45+offset0));
int t0_0_11 = rt0_0_11 * (2 + (input[i] > (46+offset0)));
t0_0_11 += (1 - rt0_0_11) * (input[i] > (44+offset0));

int rt0_0_12 = (input[i] > (49+offset0));
int t0_0_12 = rt0_0_12 * (2 + (input[i] > (50+offset0)));
t0_0_12 += (1 - rt0_0_12) * (input[i] > (48+offset0));

int rt0_0_13 = (input[i] > (53+offset0));
int t0_0_13 = rt0_0_13 * (2 + (input[i] > (54+offset0)));
t0_0_13 += (1 - rt0_0_13) * (input[i] > (52+offset0));

int rt0_0_14 = (input[i] > (57+offset0));
int t0_0_14 = rt0_0_14 * (2 + (input[i] > (58+offset0)));
t0_0_14 += (1 - rt0_0_14) * (input[i] > (56+offset0));

int rt0_0_15 = (input[i] > (61+offset0));
int t0_0_15 = rt0_0_15 * (2 + (input[i] > (62+offset0)));
t0_0_15 += (1 - rt0_0_15) * (input[i] > (60+offset0));

int rt0_0_16 = (input[i] > (65+offset0));
int t0_0_16 = rt0_0_16 * (2 + (input[i] > (66+offset0)));
t0_0_16 += (1 - rt0_0_16) * (input[i] > (64+offset0));

int rt0_0_17 = (input[i] > (69+offset0));
int t0_0_17 = rt0_0_17 * (2 + (input[i] > (70+offset0)));
t0_0_17 += (1 - rt0_0_17) * (input[i] > (68+offset0));

int rt0_0_18 = (input[i] > (73+offset0));
int t0_0_18 = rt0_0_18 * (2 + (input[i] > (74+offset0)));
t0_0_18 += (1 - rt0_0_18) * (input[i] > (72+offset0));

int rt0_0_19 = (input[i] > (77+offset0));
int t0_0_19 = rt0_0_19 * (2 + (input[i] > (78+offset0)));
t0_0_19 += (1 - rt0_0_19) * (input[i] > (76+offset0));

int rt0_0_20 = (input[i] > (81+offset0));
int t0_0_20 = rt0_0_20 * (2 + (input[i] > (82+offset0)));
t0_0_20 += (1 - rt0_0_20) * (input[i] > (80+offset0));

int rt0_0_21 = (input[i] > (85+offset0));
int t0_0_21 = rt0_0_21 * (2 + (input[i] > (86+offset0)));
t0_0_21 += (1 - rt0_0_21) * (input[i] > (84+offset0));

int rt0_0_22 = (input[i] > (89+offset0));
int t0_0_22 = rt0_0_22 * (2 + (input[i] > (90+offset0)));
t0_0_22 += (1 - rt0_0_22) * (input[i] > (88+offset0));

int rt0_0_23 = (input[i] > (93+offset0));
int t0_0_23 = rt0_0_23 * (2 + (input[i] > (94+offset0)));
t0_0_23 += (1 - rt0_0_23) * (input[i] > (92+offset0));

int rt0_0_24 = (input[i] > (97+offset0));
int t0_0_24 = rt0_0_24 * (2 + (input[i] > (98+offset0)));
t0_0_24 += (1 - rt0_0_24) * (input[i] > (96+offset0));

int rt0_0_25 = (input[i] > (101+offset0));
int t0_0_25 = rt0_0_25 * (2 + (input[i] > (102+offset0)));
t0_0_25 += (1 - rt0_0_25) * (input[i] > (100+offset0));

int rt0_0_26 = (input[i] > (105+offset0));
int t0_0_26 = rt0_0_26 * (2 + (input[i] > (106+offset0)));
t0_0_26 += (1 - rt0_0_26) * (input[i] > (104+offset0));

int rt0_0_27 = (input[i] > (109+offset0));
int t0_0_27 = rt0_0_27 * (2 + (input[i] > (110+offset0)));
t0_0_27 += (1 - rt0_0_27) * (input[i] > (108+offset0));

int rt0_0_28 = (input[i] > (113+offset0));
int t0_0_28 = rt0_0_28 * (2 + (input[i] > (114+offset0)));
t0_0_28 += (1 - rt0_0_28) * (input[i] > (112+offset0));

int rt0_0_29 = (input[i] > (117+offset0));
int t0_0_29 = rt0_0_29 * (2 + (input[i] > (118+offset0)));
t0_0_29 += (1 - rt0_0_29) * (input[i] > (116+offset0));

int rt0_0_30 = (input[i] > (121+offset0));
int t0_0_30 = rt0_0_30 * (2 + (input[i] > (122+offset0)));
t0_0_30 += (1 - rt0_0_30) * (input[i] > (120+offset0));

int rt0_0_31 = (input[i] > (125+offset0));
int t0_0_31 = rt0_0_31 * (2 + (input[i] > (126+offset0)));
t0_0_31 += (1 - rt0_0_31) * (input[i] > (124+offset0));

int rt0_0_32 = (input[i] > (129+offset0));
int t0_0_32 = rt0_0_32 * (2 + (input[i] > (130+offset0)));
t0_0_32 += (1 - rt0_0_32) * (input[i] > (128+offset0));

int rt0_0_33 = (input[i] > (133+offset0));
int t0_0_33 = rt0_0_33 * (2 + (input[i] > (134+offset0)));
t0_0_33 += (1 - rt0_0_33) * (input[i] > (132+offset0));

int rt0_0_34 = (input[i] > (137+offset0));
int t0_0_34 = rt0_0_34 * (2 + (input[i] > (138+offset0)));
t0_0_34 += (1 - rt0_0_34) * (input[i] > (136+offset0));

int rt0_0_35 = (input[i] > (141+offset0));
int t0_0_35 = rt0_0_35 * (2 + (input[i] > (142+offset0)));
t0_0_35 += (1 - rt0_0_35) * (input[i] > (140+offset0));

int rt0_0_36 = (input[i] > (145+offset0));
int t0_0_36 = rt0_0_36 * (2 + (input[i] > (146+offset0)));
t0_0_36 += (1 - rt0_0_36) * (input[i] > (144+offset0));

int rt0_0_37 = (input[i] > (149+offset0));
int t0_0_37 = rt0_0_37 * (2 + (input[i] > (150+offset0)));
t0_0_37 += (1 - rt0_0_37) * (input[i] > (148+offset0));

int rt0_0_38 = (input[i] > (153+offset0));
int t0_0_38 = rt0_0_38 * (2 + (input[i] > (154+offset0)));
t0_0_38 += (1 - rt0_0_38) * (input[i] > (152+offset0));

int rt0_0_39 = (input[i] > (157+offset0));
int t0_0_39 = rt0_0_39 * (2 + (input[i] > (158+offset0)));
t0_0_39 += (1 - rt0_0_39) * (input[i] > (156+offset0));

int rt0_0_40 = (input[i] > (161+offset0));
int t0_0_40 = rt0_0_40 * (2 + (input[i] > (162+offset0)));
t0_0_40 += (1 - rt0_0_40) * (input[i] > (160+offset0));

int rt0_0_41 = (input[i] > (165+offset0));
int t0_0_41 = rt0_0_41 * (2 + (input[i] > (166+offset0)));
t0_0_41 += (1 - rt0_0_41) * (input[i] > (164+offset0));

int rt0_0_42 = (input[i] > (169+offset0));
int t0_0_42 = rt0_0_42 * (2 + (input[i] > (170+offset0)));
t0_0_42 += (1 - rt0_0_42) * (input[i] > (168+offset0));

int rt0_0_43 = (input[i] > (173+offset0));
int t0_0_43 = rt0_0_43 * (2 + (input[i] > (174+offset0)));
t0_0_43 += (1 - rt0_0_43) * (input[i] > (172+offset0));

int rt0_0_44 = (input[i] > (177+offset0));
int t0_0_44 = rt0_0_44 * (2 + (input[i] > (178+offset0)));
t0_0_44 += (1 - rt0_0_44) * (input[i] > (176+offset0));

int rt0_0_45 = (input[i] > (181+offset0));
int t0_0_45 = rt0_0_45 * (2 + (input[i] > (182+offset0)));
t0_0_45 += (1 - rt0_0_45) * (input[i] > (180+offset0));

int rt0_0_46 = (input[i] > (185+offset0));
int t0_0_46 = rt0_0_46 * (2 + (input[i] > (186+offset0)));
t0_0_46 += (1 - rt0_0_46) * (input[i] > (184+offset0));

int rt0_0_47 = (input[i] > (189+offset0));
int t0_0_47 = rt0_0_47 * (2 + (input[i] > (190+offset0)));
t0_0_47 += (1 - rt0_0_47) * (input[i] > (188+offset0));

int rt0_0_48 = (input[i] > (193+offset0));
int t0_0_48 = rt0_0_48 * (2 + (input[i] > (194+offset0)));
t0_0_48 += (1 - rt0_0_48) * (input[i] > (192+offset0));

int rt0_0_49 = (input[i] > (197+offset0));
int t0_0_49 = rt0_0_49 * (2 + (input[i] > (198+offset0)));
t0_0_49 += (1 - rt0_0_49) * (input[i] > (196+offset0));

int rt0_0_50 = (input[i] > (201+offset0));
int t0_0_50 = rt0_0_50 * (2 + (input[i] > (202+offset0)));
t0_0_50 += (1 - rt0_0_50) * (input[i] > (200+offset0));

int rt0_0_51 = (input[i] > (205+offset0));
int t0_0_51 = rt0_0_51 * (2 + (input[i] > (206+offset0)));
t0_0_51 += (1 - rt0_0_51) * (input[i] > (204+offset0));

int rt0_0_52 = (input[i] > (209+offset0));
int t0_0_52 = rt0_0_52 * (2 + (input[i] > (210+offset0)));
t0_0_52 += (1 - rt0_0_52) * (input[i] > (208+offset0));

int rt0_0_53 = (input[i] > (213+offset0));
int t0_0_53 = rt0_0_53 * (2 + (input[i] > (214+offset0)));
t0_0_53 += (1 - rt0_0_53) * (input[i] > (212+offset0));

int rt0_0_54 = (input[i] > (217+offset0));
int t0_0_54 = rt0_0_54 * (2 + (input[i] > (218+offset0)));
t0_0_54 += (1 - rt0_0_54) * (input[i] > (216+offset0));

int rt0_0_55 = (input[i] > (221+offset0));
int t0_0_55 = rt0_0_55 * (2 + (input[i] > (222+offset0)));
t0_0_55 += (1 - rt0_0_55) * (input[i] > (220+offset0));

int rt0_0_56 = (input[i] > (225+offset0));
int t0_0_56 = rt0_0_56 * (2 + (input[i] > (226+offset0)));
t0_0_56 += (1 - rt0_0_56) * (input[i] > (224+offset0));

int rt0_0_57 = (input[i] > (229+offset0));
int t0_0_57 = rt0_0_57 * (2 + (input[i] > (230+offset0)));
t0_0_57 += (1 - rt0_0_57) * (input[i] > (228+offset0));

int rt0_0_58 = (input[i] > (233+offset0));
int t0_0_58 = rt0_0_58 * (2 + (input[i] > (234+offset0)));
t0_0_58 += (1 - rt0_0_58) * (input[i] > (232+offset0));

int rt0_0_59 = (input[i] > (237+offset0));
int t0_0_59 = rt0_0_59 * (2 + (input[i] > (238+offset0)));
t0_0_59 += (1 - rt0_0_59) * (input[i] > (236+offset0));

int rt0_0_60 = (input[i] > (241+offset0));
int t0_0_60 = rt0_0_60 * (2 + (input[i] > (242+offset0)));
t0_0_60 += (1 - rt0_0_60) * (input[i] > (240+offset0));

int rt0_0_61 = (input[i] > (245+offset0));
int t0_0_61 = rt0_0_61 * (2 + (input[i] > (246+offset0)));
t0_0_61 += (1 - rt0_0_61) * (input[i] > (244+offset0));

int rt0_0_62 = (input[i] > (249+offset0));
int t0_0_62 = rt0_0_62 * (2 + (input[i] > (250+offset0)));
t0_0_62 += (1 - rt0_0_62) * (input[i] > (248+offset0));

int rt0_0_63 = (input[i] > (253+offset0));
int t0_0_63 = rt0_0_63 * (2 + (input[i] > (254+offset0)));
t0_0_63 += (1 - rt0_0_63) * (input[i] > (252+offset0));

int rt0_0_64 = (input[i] > (257+offset0));
int t0_0_64 = rt0_0_64 * (2 + (input[i] > (258+offset0)));
t0_0_64 += (1 - rt0_0_64) * (input[i] > (256+offset0));

int rt0_0_65 = (input[i] > (261+offset0));
int t0_0_65 = rt0_0_65 * (2 + (input[i] > (262+offset0)));
t0_0_65 += (1 - rt0_0_65) * (input[i] > (260+offset0));

int rt0_0_66 = (input[i] > (265+offset0));
int t0_0_66 = rt0_0_66 * (2 + (input[i] > (266+offset0)));
t0_0_66 += (1 - rt0_0_66) * (input[i] > (264+offset0));

int rt0_0_67 = (input[i] > (269+offset0));
int t0_0_67 = rt0_0_67 * (2 + (input[i] > (270+offset0)));
t0_0_67 += (1 - rt0_0_67) * (input[i] > (268+offset0));

int rt0_0_68 = (input[i] > (273+offset0));
int t0_0_68 = rt0_0_68 * (2 + (input[i] > (274+offset0)));
t0_0_68 += (1 - rt0_0_68) * (input[i] > (272+offset0));

int rt0_0_69 = (input[i] > (277+offset0));
int t0_0_69 = rt0_0_69 * (2 + (input[i] > (278+offset0)));
t0_0_69 += (1 - rt0_0_69) * (input[i] > (276+offset0));

int rt0_0_70 = (input[i] > (281+offset0));
int t0_0_70 = rt0_0_70 * (2 + (input[i] > (282+offset0)));
t0_0_70 += (1 - rt0_0_70) * (input[i] > (280+offset0));

int rt0_0_71 = (input[i] > (285+offset0));
int t0_0_71 = rt0_0_71 * (2 + (input[i] > (286+offset0)));
t0_0_71 += (1 - rt0_0_71) * (input[i] > (284+offset0));

int rt0_0_72 = (input[i] > (289+offset0));
int t0_0_72 = rt0_0_72 * (2 + (input[i] > (290+offset0)));
t0_0_72 += (1 - rt0_0_72) * (input[i] > (288+offset0));

int rt0_0_73 = (input[i] > (293+offset0));
int t0_0_73 = rt0_0_73 * (2 + (input[i] > (294+offset0)));
t0_0_73 += (1 - rt0_0_73) * (input[i] > (292+offset0));

int rt0_0_74 = (input[i] > (297+offset0));
int t0_0_74 = rt0_0_74 * (2 + (input[i] > (298+offset0)));
t0_0_74 += (1 - rt0_0_74) * (input[i] > (296+offset0));

int rt0_0_75 = (input[i] > (301+offset0));
int t0_0_75 = rt0_0_75 * (2 + (input[i] > (302+offset0)));
t0_0_75 += (1 - rt0_0_75) * (input[i] > (300+offset0));

int rt0_0_76 = (input[i] > (305+offset0));
int t0_0_76 = rt0_0_76 * (2 + (input[i] > (306+offset0)));
t0_0_76 += (1 - rt0_0_76) * (input[i] > (304+offset0));

int rt0_0_77 = (input[i] > (309+offset0));
int t0_0_77 = rt0_0_77 * (2 + (input[i] > (310+offset0)));
t0_0_77 += (1 - rt0_0_77) * (input[i] > (308+offset0));

int rt0_0_78 = (input[i] > (313+offset0));
int t0_0_78 = rt0_0_78 * (2 + (input[i] > (314+offset0)));
t0_0_78 += (1 - rt0_0_78) * (input[i] > (312+offset0));

int rt0_0_79 = (input[i] > (317+offset0));
int t0_0_79 = rt0_0_79 * (2 + (input[i] > (318+offset0)));
t0_0_79 += (1 - rt0_0_79) * (input[i] > (316+offset0));

int rt0_0_80 = (input[i] > (321+offset0));
int t0_0_80 = rt0_0_80 * (2 + (input[i] > (322+offset0)));
t0_0_80 += (1 - rt0_0_80) * (input[i] > (320+offset0));

int rt0_0_81 = (input[i] > (325+offset0));
int t0_0_81 = rt0_0_81 * (2 + (input[i] > (326+offset0)));
t0_0_81 += (1 - rt0_0_81) * (input[i] > (324+offset0));

int rt0_0_82 = (input[i] > (329+offset0));
int t0_0_82 = rt0_0_82 * (2 + (input[i] > (330+offset0)));
t0_0_82 += (1 - rt0_0_82) * (input[i] > (328+offset0));

int rt0_0_83 = (input[i] > (333+offset0));
int t0_0_83 = rt0_0_83 * (2 + (input[i] > (334+offset0)));
t0_0_83 += (1 - rt0_0_83) * (input[i] > (332+offset0));

int rt0_0_84 = (input[i] > (337+offset0));
int t0_0_84 = rt0_0_84 * (2 + (input[i] > (338+offset0)));
t0_0_84 += (1 - rt0_0_84) * (input[i] > (336+offset0));

int rt0_0_85 = (input[i] > (341+offset0));
int t0_0_85 = rt0_0_85 * (2 + (input[i] > (342+offset0)));
t0_0_85 += (1 - rt0_0_85) * (input[i] > (340+offset0));

int rt0_0_86 = (input[i] > (345+offset0));
int t0_0_86 = rt0_0_86 * (2 + (input[i] > (346+offset0)));
t0_0_86 += (1 - rt0_0_86) * (input[i] > (344+offset0));

int rt0_0_87 = (input[i] > (349+offset0));
int t0_0_87 = rt0_0_87 * (2 + (input[i] > (350+offset0)));
t0_0_87 += (1 - rt0_0_87) * (input[i] > (348+offset0));

int rt0_0_88 = (input[i] > (353+offset0));
int t0_0_88 = rt0_0_88 * (2 + (input[i] > (354+offset0)));
t0_0_88 += (1 - rt0_0_88) * (input[i] > (352+offset0));

int rt0_0_89 = (input[i] > (357+offset0));
int t0_0_89 = rt0_0_89 * (2 + (input[i] > (358+offset0)));
t0_0_89 += (1 - rt0_0_89) * (input[i] > (356+offset0));

int rt0_0_90 = (input[i] > (361+offset0));
int t0_0_90 = rt0_0_90 * (2 + (input[i] > (362+offset0)));
t0_0_90 += (1 - rt0_0_90) * (input[i] > (360+offset0));

int rt0_0_91 = (input[i] > (365+offset0));
int t0_0_91 = rt0_0_91 * (2 + (input[i] > (366+offset0)));
t0_0_91 += (1 - rt0_0_91) * (input[i] > (364+offset0));

int rt0_0_92 = (input[i] > (369+offset0));
int t0_0_92 = rt0_0_92 * (2 + (input[i] > (370+offset0)));
t0_0_92 += (1 - rt0_0_92) * (input[i] > (368+offset0));

int rt0_0_93 = (input[i] > (373+offset0));
int t0_0_93 = rt0_0_93 * (2 + (input[i] > (374+offset0)));
t0_0_93 += (1 - rt0_0_93) * (input[i] > (372+offset0));

int rt0_0_94 = (input[i] > (377+offset0));
int t0_0_94 = rt0_0_94 * (2 + (input[i] > (378+offset0)));
t0_0_94 += (1 - rt0_0_94) * (input[i] > (376+offset0));

int rt0_0_95 = (input[i] > (381+offset0));
int t0_0_95 = rt0_0_95 * (2 + (input[i] > (382+offset0)));
t0_0_95 += (1 - rt0_0_95) * (input[i] > (380+offset0));

int rt0_0_96 = (input[i] > (385+offset0));
int t0_0_96 = rt0_0_96 * (2 + (input[i] > (386+offset0)));
t0_0_96 += (1 - rt0_0_96) * (input[i] > (384+offset0));

int rt0_0_97 = (input[i] > (389+offset0));
int t0_0_97 = rt0_0_97 * (2 + (input[i] > (390+offset0)));
t0_0_97 += (1 - rt0_0_97) * (input[i] > (388+offset0));

int rt0_0_98 = (input[i] > (393+offset0));
int t0_0_98 = rt0_0_98 * (2 + (input[i] > (394+offset0)));
t0_0_98 += (1 - rt0_0_98) * (input[i] > (392+offset0));

int rt0_0_99 = (input[i] > (397+offset0));
int t0_0_99 = rt0_0_99 * (2 + (input[i] > (398+offset0)));
t0_0_99 += (1 - rt0_0_99) * (input[i] > (396+offset0));

int rt0_0_100 = (input[i] > (401+offset0));
int t0_0_100 = rt0_0_100 * (2 + (input[i] > (402+offset0)));
t0_0_100 += (1 - rt0_0_100) * (input[i] > (400+offset0));

int rt0_0_101 = (input[i] > (405+offset0));
int t0_0_101 = rt0_0_101 * (2 + (input[i] > (406+offset0)));
t0_0_101 += (1 - rt0_0_101) * (input[i] > (404+offset0));

int rt0_0_102 = (input[i] > (409+offset0));
int t0_0_102 = rt0_0_102 * (2 + (input[i] > (410+offset0)));
t0_0_102 += (1 - rt0_0_102) * (input[i] > (408+offset0));

int rt0_0_103 = (input[i] > (413+offset0));
int t0_0_103 = rt0_0_103 * (2 + (input[i] > (414+offset0)));
t0_0_103 += (1 - rt0_0_103) * (input[i] > (412+offset0));

int rt0_0_104 = (input[i] > (417+offset0));
int t0_0_104 = rt0_0_104 * (2 + (input[i] > (418+offset0)));
t0_0_104 += (1 - rt0_0_104) * (input[i] > (416+offset0));

int rt0_0_105 = (input[i] > (421+offset0));
int t0_0_105 = rt0_0_105 * (2 + (input[i] > (422+offset0)));
t0_0_105 += (1 - rt0_0_105) * (input[i] > (420+offset0));

int rt0_0_106 = (input[i] > (425+offset0));
int t0_0_106 = rt0_0_106 * (2 + (input[i] > (426+offset0)));
t0_0_106 += (1 - rt0_0_106) * (input[i] > (424+offset0));

int rt0_0_107 = (input[i] > (429+offset0));
int t0_0_107 = rt0_0_107 * (2 + (input[i] > (430+offset0)));
t0_0_107 += (1 - rt0_0_107) * (input[i] > (428+offset0));

int rt0_0_108 = (input[i] > (433+offset0));
int t0_0_108 = rt0_0_108 * (2 + (input[i] > (434+offset0)));
t0_0_108 += (1 - rt0_0_108) * (input[i] > (432+offset0));

int rt0_0_109 = (input[i] > (437+offset0));
int t0_0_109 = rt0_0_109 * (2 + (input[i] > (438+offset0)));
t0_0_109 += (1 - rt0_0_109) * (input[i] > (436+offset0));

int rt0_0_110 = (input[i] > (441+offset0));
int t0_0_110 = rt0_0_110 * (2 + (input[i] > (442+offset0)));
t0_0_110 += (1 - rt0_0_110) * (input[i] > (440+offset0));

int rt0_0_111 = (input[i] > (445+offset0));
int t0_0_111 = rt0_0_111 * (2 + (input[i] > (446+offset0)));
t0_0_111 += (1 - rt0_0_111) * (input[i] > (444+offset0));

int rt0_0_112 = (input[i] > (449+offset0));
int t0_0_112 = rt0_0_112 * (2 + (input[i] > (450+offset0)));
t0_0_112 += (1 - rt0_0_112) * (input[i] > (448+offset0));

int rt0_0_113 = (input[i] > (453+offset0));
int t0_0_113 = rt0_0_113 * (2 + (input[i] > (454+offset0)));
t0_0_113 += (1 - rt0_0_113) * (input[i] > (452+offset0));

int rt0_0_114 = (input[i] > (457+offset0));
int t0_0_114 = rt0_0_114 * (2 + (input[i] > (458+offset0)));
t0_0_114 += (1 - rt0_0_114) * (input[i] > (456+offset0));

int rt0_0_115 = (input[i] > (461+offset0));
int t0_0_115 = rt0_0_115 * (2 + (input[i] > (462+offset0)));
t0_0_115 += (1 - rt0_0_115) * (input[i] > (460+offset0));

int rt0_0_116 = (input[i] > (465+offset0));
int t0_0_116 = rt0_0_116 * (2 + (input[i] > (466+offset0)));
t0_0_116 += (1 - rt0_0_116) * (input[i] > (464+offset0));

int rt0_0_117 = (input[i] > (469+offset0));
int t0_0_117 = rt0_0_117 * (2 + (input[i] > (470+offset0)));
t0_0_117 += (1 - rt0_0_117) * (input[i] > (468+offset0));

int rt0_0_118 = (input[i] > (473+offset0));
int t0_0_118 = rt0_0_118 * (2 + (input[i] > (474+offset0)));
t0_0_118 += (1 - rt0_0_118) * (input[i] > (472+offset0));

int rt0_0_119 = (input[i] > (477+offset0));
int t0_0_119 = rt0_0_119 * (2 + (input[i] > (478+offset0)));
t0_0_119 += (1 - rt0_0_119) * (input[i] > (476+offset0));

int rt0_0_120 = (input[i] > (481+offset0));
int t0_0_120 = rt0_0_120 * (2 + (input[i] > (482+offset0)));
t0_0_120 += (1 - rt0_0_120) * (input[i] > (480+offset0));

int rt0_0_121 = (input[i] > (485+offset0));
int t0_0_121 = rt0_0_121 * (2 + (input[i] > (486+offset0)));
t0_0_121 += (1 - rt0_0_121) * (input[i] > (484+offset0));

int rt0_0_122 = (input[i] > (489+offset0));
int t0_0_122 = rt0_0_122 * (2 + (input[i] > (490+offset0)));
t0_0_122 += (1 - rt0_0_122) * (input[i] > (488+offset0));

int rt0_0_123 = (input[i] > (493+offset0));
int t0_0_123 = rt0_0_123 * (2 + (input[i] > (494+offset0)));
t0_0_123 += (1 - rt0_0_123) * (input[i] > (492+offset0));

int rt0_0_124 = (input[i] > (497+offset0));
int t0_0_124 = rt0_0_124 * (2 + (input[i] > (498+offset0)));
t0_0_124 += (1 - rt0_0_124) * (input[i] > (496+offset0));

int rt0_0_125 = (input[i] > (501+offset0));
int t0_0_125 = rt0_0_125 * (2 + (input[i] > (502+offset0)));
t0_0_125 += (1 - rt0_0_125) * (input[i] > (500+offset0));

int rt0_0_126 = (input[i] > (505+offset0));
int t0_0_126 = rt0_0_126 * (2 + (input[i] > (506+offset0)));
t0_0_126 += (1 - rt0_0_126) * (input[i] > (504+offset0));

int rt0_0_127 = (input[i] > (509+offset0));
int t0_0_127 = rt0_0_127 * (2 + (input[i] > (510+offset0)));
t0_0_127 += (1 - rt0_0_127) * (input[i] > (508+offset0));

int rt0_0_128 = (input[i] > (513+offset0));
int t0_0_128 = rt0_0_128 * (2 + (input[i] > (514+offset0)));
t0_0_128 += (1 - rt0_0_128) * (input[i] > (512+offset0));

int rt0_0_129 = (input[i] > (517+offset0));
int t0_0_129 = rt0_0_129 * (2 + (input[i] > (518+offset0)));
t0_0_129 += (1 - rt0_0_129) * (input[i] > (516+offset0));

int rt0_0_130 = (input[i] > (521+offset0));
int t0_0_130 = rt0_0_130 * (2 + (input[i] > (522+offset0)));
t0_0_130 += (1 - rt0_0_130) * (input[i] > (520+offset0));

int rt0_0_131 = (input[i] > (525+offset0));
int t0_0_131 = rt0_0_131 * (2 + (input[i] > (526+offset0)));
t0_0_131 += (1 - rt0_0_131) * (input[i] > (524+offset0));

int rt0_0_132 = (input[i] > (529+offset0));
int t0_0_132 = rt0_0_132 * (2 + (input[i] > (530+offset0)));
t0_0_132 += (1 - rt0_0_132) * (input[i] > (528+offset0));

int rt0_0_133 = (input[i] > (533+offset0));
int t0_0_133 = rt0_0_133 * (2 + (input[i] > (534+offset0)));
t0_0_133 += (1 - rt0_0_133) * (input[i] > (532+offset0));

int rt0_0_134 = (input[i] > (537+offset0));
int t0_0_134 = rt0_0_134 * (2 + (input[i] > (538+offset0)));
t0_0_134 += (1 - rt0_0_134) * (input[i] > (536+offset0));

int rt0_0_135 = (input[i] > (541+offset0));
int t0_0_135 = rt0_0_135 * (2 + (input[i] > (542+offset0)));
t0_0_135 += (1 - rt0_0_135) * (input[i] > (540+offset0));

int rt0_0_136 = (input[i] > (545+offset0));
int t0_0_136 = rt0_0_136 * (2 + (input[i] > (546+offset0)));
t0_0_136 += (1 - rt0_0_136) * (input[i] > (544+offset0));

int rt0_0_137 = (input[i] > (549+offset0));
int t0_0_137 = rt0_0_137 * (2 + (input[i] > (550+offset0)));
t0_0_137 += (1 - rt0_0_137) * (input[i] > (548+offset0));

int rt0_0_138 = (input[i] > (553+offset0));
int t0_0_138 = rt0_0_138 * (2 + (input[i] > (554+offset0)));
t0_0_138 += (1 - rt0_0_138) * (input[i] > (552+offset0));

int rt0_0_139 = (input[i] > (557+offset0));
int t0_0_139 = rt0_0_139 * (2 + (input[i] > (558+offset0)));
t0_0_139 += (1 - rt0_0_139) * (input[i] > (556+offset0));

int rt0_0_140 = (input[i] > (561+offset0));
int t0_0_140 = rt0_0_140 * (2 + (input[i] > (562+offset0)));
t0_0_140 += (1 - rt0_0_140) * (input[i] > (560+offset0));

int rt0_0_141 = (input[i] > (565+offset0));
int t0_0_141 = rt0_0_141 * (2 + (input[i] > (566+offset0)));
t0_0_141 += (1 - rt0_0_141) * (input[i] > (564+offset0));

int rt0_0_142 = (input[i] > (569+offset0));
int t0_0_142 = rt0_0_142 * (2 + (input[i] > (570+offset0)));
t0_0_142 += (1 - rt0_0_142) * (input[i] > (568+offset0));

int rt0_0_143 = (input[i] > (573+offset0));
int t0_0_143 = rt0_0_143 * (2 + (input[i] > (574+offset0)));
t0_0_143 += (1 - rt0_0_143) * (input[i] > (572+offset0));

int rt0_0_144 = (input[i] > (577+offset0));
int t0_0_144 = rt0_0_144 * (2 + (input[i] > (578+offset0)));
t0_0_144 += (1 - rt0_0_144) * (input[i] > (576+offset0));

int rt0_0_145 = (input[i] > (581+offset0));
int t0_0_145 = rt0_0_145 * (2 + (input[i] > (582+offset0)));
t0_0_145 += (1 - rt0_0_145) * (input[i] > (580+offset0));

int rt0_0_146 = (input[i] > (585+offset0));
int t0_0_146 = rt0_0_146 * (2 + (input[i] > (586+offset0)));
t0_0_146 += (1 - rt0_0_146) * (input[i] > (584+offset0));

int rt0_0_147 = (input[i] > (589+offset0));
int t0_0_147 = rt0_0_147 * (2 + (input[i] > (590+offset0)));
t0_0_147 += (1 - rt0_0_147) * (input[i] > (588+offset0));

int rt0_0_148 = (input[i] > (593+offset0));
int t0_0_148 = rt0_0_148 * (2 + (input[i] > (594+offset0)));
t0_0_148 += (1 - rt0_0_148) * (input[i] > (592+offset0));

int rt0_0_149 = (input[i] > (597+offset0));
int t0_0_149 = rt0_0_149 * (2 + (input[i] > (598+offset0)));
t0_0_149 += (1 - rt0_0_149) * (input[i] > (596+offset0));

int rt0_0_150 = (input[i] > (601+offset0));
int t0_0_150 = rt0_0_150 * (2 + (input[i] > (602+offset0)));
t0_0_150 += (1 - rt0_0_150) * (input[i] > (600+offset0));

int rt0_0_151 = (input[i] > (605+offset0));
int t0_0_151 = rt0_0_151 * (2 + (input[i] > (606+offset0)));
t0_0_151 += (1 - rt0_0_151) * (input[i] > (604+offset0));

int rt0_0_152 = (input[i] > (609+offset0));
int t0_0_152 = rt0_0_152 * (2 + (input[i] > (610+offset0)));
t0_0_152 += (1 - rt0_0_152) * (input[i] > (608+offset0));

int rt0_0_153 = (input[i] > (613+offset0));
int t0_0_153 = rt0_0_153 * (2 + (input[i] > (614+offset0)));
t0_0_153 += (1 - rt0_0_153) * (input[i] > (612+offset0));

int rt0_0_154 = (input[i] > (617+offset0));
int t0_0_154 = rt0_0_154 * (2 + (input[i] > (618+offset0)));
t0_0_154 += (1 - rt0_0_154) * (input[i] > (616+offset0));

int rt0_0_155 = (input[i] > (621+offset0));
int t0_0_155 = rt0_0_155 * (2 + (input[i] > (622+offset0)));
t0_0_155 += (1 - rt0_0_155) * (input[i] > (620+offset0));

int rt0_0_156 = (input[i] > (625+offset0));
int t0_0_156 = rt0_0_156 * (2 + (input[i] > (626+offset0)));
t0_0_156 += (1 - rt0_0_156) * (input[i] > (624+offset0));

int rt0_0_157 = (input[i] > (629+offset0));
int t0_0_157 = rt0_0_157 * (2 + (input[i] > (630+offset0)));
t0_0_157 += (1 - rt0_0_157) * (input[i] > (628+offset0));

int rt0_0_158 = (input[i] > (633+offset0));
int t0_0_158 = rt0_0_158 * (2 + (input[i] > (634+offset0)));
t0_0_158 += (1 - rt0_0_158) * (input[i] > (632+offset0));

int rt0_0_159 = (input[i] > (637+offset0));
int t0_0_159 = rt0_0_159 * (2 + (input[i] > (638+offset0)));
t0_0_159 += (1 - rt0_0_159) * (input[i] > (636+offset0));

int rt0_0_160 = (input[i] > (641+offset0));
int t0_0_160 = rt0_0_160 * (2 + (input[i] > (642+offset0)));
t0_0_160 += (1 - rt0_0_160) * (input[i] > (640+offset0));

int rt0_0_161 = (input[i] > (645+offset0));
int t0_0_161 = rt0_0_161 * (2 + (input[i] > (646+offset0)));
t0_0_161 += (1 - rt0_0_161) * (input[i] > (644+offset0));

int rt0_0_162 = (input[i] > (649+offset0));
int t0_0_162 = rt0_0_162 * (2 + (input[i] > (650+offset0)));
t0_0_162 += (1 - rt0_0_162) * (input[i] > (648+offset0));

int rt0_0_163 = (input[i] > (653+offset0));
int t0_0_163 = rt0_0_163 * (2 + (input[i] > (654+offset0)));
t0_0_163 += (1 - rt0_0_163) * (input[i] > (652+offset0));

int rt0_0_164 = (input[i] > (657+offset0));
int t0_0_164 = rt0_0_164 * (2 + (input[i] > (658+offset0)));
t0_0_164 += (1 - rt0_0_164) * (input[i] > (656+offset0));

int rt0_0_165 = (input[i] > (661+offset0));
int t0_0_165 = rt0_0_165 * (2 + (input[i] > (662+offset0)));
t0_0_165 += (1 - rt0_0_165) * (input[i] > (660+offset0));

int rt0_0_166 = (input[i] > (665+offset0));
int t0_0_166 = rt0_0_166 * (2 + (input[i] > (666+offset0)));
t0_0_166 += (1 - rt0_0_166) * (input[i] > (664+offset0));

int rt0_0_167 = (input[i] > (669+offset0));
int t0_0_167 = rt0_0_167 * (2 + (input[i] > (670+offset0)));
t0_0_167 += (1 - rt0_0_167) * (input[i] > (668+offset0));

int rt0_0_168 = (input[i] > (673+offset0));
int t0_0_168 = rt0_0_168 * (2 + (input[i] > (674+offset0)));
t0_0_168 += (1 - rt0_0_168) * (input[i] > (672+offset0));

int rt0_0_169 = (input[i] > (677+offset0));
int t0_0_169 = rt0_0_169 * (2 + (input[i] > (678+offset0)));
t0_0_169 += (1 - rt0_0_169) * (input[i] > (676+offset0));

int rt0_0_170 = (input[i] > (681+offset0));
int t0_0_170 = rt0_0_170 * (2 + (input[i] > (682+offset0)));
t0_0_170 += (1 - rt0_0_170) * (input[i] > (680+offset0));

int rt0_0_171 = (input[i] > (685+offset0));
int t0_0_171 = rt0_0_171 * (2 + (input[i] > (686+offset0)));
t0_0_171 += (1 - rt0_0_171) * (input[i] > (684+offset0));

int rt0_0_172 = (input[i] > (689+offset0));
int t0_0_172 = rt0_0_172 * (2 + (input[i] > (690+offset0)));
t0_0_172 += (1 - rt0_0_172) * (input[i] > (688+offset0));

int rt0_0_173 = (input[i] > (693+offset0));
int t0_0_173 = rt0_0_173 * (2 + (input[i] > (694+offset0)));
t0_0_173 += (1 - rt0_0_173) * (input[i] > (692+offset0));

int rt0_0_174 = (input[i] > (697+offset0));
int t0_0_174 = rt0_0_174 * (2 + (input[i] > (698+offset0)));
t0_0_174 += (1 - rt0_0_174) * (input[i] > (696+offset0));

int rt0_0_175 = (input[i] > (701+offset0));
int t0_0_175 = rt0_0_175 * (2 + (input[i] > (702+offset0)));
t0_0_175 += (1 - rt0_0_175) * (input[i] > (700+offset0));

int rt0_0_176 = (input[i] > (705+offset0));
int t0_0_176 = rt0_0_176 * (2 + (input[i] > (706+offset0)));
t0_0_176 += (1 - rt0_0_176) * (input[i] > (704+offset0));

int rt0_0_177 = (input[i] > (709+offset0));
int t0_0_177 = rt0_0_177 * (2 + (input[i] > (710+offset0)));
t0_0_177 += (1 - rt0_0_177) * (input[i] > (708+offset0));

int rt0_0_178 = (input[i] > (713+offset0));
int t0_0_178 = rt0_0_178 * (2 + (input[i] > (714+offset0)));
t0_0_178 += (1 - rt0_0_178) * (input[i] > (712+offset0));

int rt0_0_179 = (input[i] > (717+offset0));
int t0_0_179 = rt0_0_179 * (2 + (input[i] > (718+offset0)));
t0_0_179 += (1 - rt0_0_179) * (input[i] > (716+offset0));

int rt0_0_180 = (input[i] > (721+offset0));
int t0_0_180 = rt0_0_180 * (2 + (input[i] > (722+offset0)));
t0_0_180 += (1 - rt0_0_180) * (input[i] > (720+offset0));

int rt0_0_181 = (input[i] > (725+offset0));
int t0_0_181 = rt0_0_181 * (2 + (input[i] > (726+offset0)));
t0_0_181 += (1 - rt0_0_181) * (input[i] > (724+offset0));

int rt0_0_182 = (input[i] > (729+offset0));
int t0_0_182 = rt0_0_182 * (2 + (input[i] > (730+offset0)));
t0_0_182 += (1 - rt0_0_182) * (input[i] > (728+offset0));

int rt0_0_183 = (input[i] > (733+offset0));
int t0_0_183 = rt0_0_183 * (2 + (input[i] > (734+offset0)));
t0_0_183 += (1 - rt0_0_183) * (input[i] > (732+offset0));

int rt0_0_184 = (input[i] > (737+offset0));
int t0_0_184 = rt0_0_184 * (2 + (input[i] > (738+offset0)));
t0_0_184 += (1 - rt0_0_184) * (input[i] > (736+offset0));

int rt0_0_185 = (input[i] > (741+offset0));
int t0_0_185 = rt0_0_185 * (2 + (input[i] > (742+offset0)));
t0_0_185 += (1 - rt0_0_185) * (input[i] > (740+offset0));

int rt0_0_186 = (input[i] > (745+offset0));
int t0_0_186 = rt0_0_186 * (2 + (input[i] > (746+offset0)));
t0_0_186 += (1 - rt0_0_186) * (input[i] > (744+offset0));

int rt0_0_187 = (input[i] > (749+offset0));
int t0_0_187 = rt0_0_187 * (2 + (input[i] > (750+offset0)));
t0_0_187 += (1 - rt0_0_187) * (input[i] > (748+offset0));

int rt0_0_188 = (input[i] > (753+offset0));
int t0_0_188 = rt0_0_188 * (2 + (input[i] > (754+offset0)));
t0_0_188 += (1 - rt0_0_188) * (input[i] > (752+offset0));

int rt0_0_189 = (input[i] > (757+offset0));
int t0_0_189 = rt0_0_189 * (2 + (input[i] > (758+offset0)));
t0_0_189 += (1 - rt0_0_189) * (input[i] > (756+offset0));

int rt0_0_190 = (input[i] > (761+offset0));
int t0_0_190 = rt0_0_190 * (2 + (input[i] > (762+offset0)));
t0_0_190 += (1 - rt0_0_190) * (input[i] > (760+offset0));

int rt0_0_191 = (input[i] > (765+offset0));
int t0_0_191 = rt0_0_191 * (2 + (input[i] > (766+offset0)));
t0_0_191 += (1 - rt0_0_191) * (input[i] > (764+offset0));

int rt0_0_192 = (input[i] > (769+offset0));
int t0_0_192 = rt0_0_192 * (2 + (input[i] > (770+offset0)));
t0_0_192 += (1 - rt0_0_192) * (input[i] > (768+offset0));

int rt0_0_193 = (input[i] > (773+offset0));
int t0_0_193 = rt0_0_193 * (2 + (input[i] > (774+offset0)));
t0_0_193 += (1 - rt0_0_193) * (input[i] > (772+offset0));

int rt0_0_194 = (input[i] > (777+offset0));
int t0_0_194 = rt0_0_194 * (2 + (input[i] > (778+offset0)));
t0_0_194 += (1 - rt0_0_194) * (input[i] > (776+offset0));

int rt0_0_195 = (input[i] > (781+offset0));
int t0_0_195 = rt0_0_195 * (2 + (input[i] > (782+offset0)));
t0_0_195 += (1 - rt0_0_195) * (input[i] > (780+offset0));

int rt0_0_196 = (input[i] > (785+offset0));
int t0_0_196 = rt0_0_196 * (2 + (input[i] > (786+offset0)));
t0_0_196 += (1 - rt0_0_196) * (input[i] > (784+offset0));

int rt0_0_197 = (input[i] > (789+offset0));
int t0_0_197 = rt0_0_197 * (2 + (input[i] > (790+offset0)));
t0_0_197 += (1 - rt0_0_197) * (input[i] > (788+offset0));

int rt0_0_198 = (input[i] > (793+offset0));
int t0_0_198 = rt0_0_198 * (2 + (input[i] > (794+offset0)));
t0_0_198 += (1 - rt0_0_198) * (input[i] > (792+offset0));

int rt0_0_199 = (input[i] > (797+offset0));
int t0_0_199 = rt0_0_199 * (2 + (input[i] > (798+offset0)));
t0_0_199 += (1 - rt0_0_199) * (input[i] > (796+offset0));

int rt0_0_200 = (input[i] > (801+offset0));
int t0_0_200 = rt0_0_200 * (2 + (input[i] > (802+offset0)));
t0_0_200 += (1 - rt0_0_200) * (input[i] > (800+offset0));

int rt0_0_201 = (input[i] > (805+offset0));
int t0_0_201 = rt0_0_201 * (2 + (input[i] > (806+offset0)));
t0_0_201 += (1 - rt0_0_201) * (input[i] > (804+offset0));

int rt0_0_202 = (input[i] > (809+offset0));
int t0_0_202 = rt0_0_202 * (2 + (input[i] > (810+offset0)));
t0_0_202 += (1 - rt0_0_202) * (input[i] > (808+offset0));

int rt0_0_203 = (input[i] > (813+offset0));
int t0_0_203 = rt0_0_203 * (2 + (input[i] > (814+offset0)));
t0_0_203 += (1 - rt0_0_203) * (input[i] > (812+offset0));

int rt0_0_204 = (input[i] > (817+offset0));
int t0_0_204 = rt0_0_204 * (2 + (input[i] > (818+offset0)));
t0_0_204 += (1 - rt0_0_204) * (input[i] > (816+offset0));

int rt0_0_205 = (input[i] > (821+offset0));
int t0_0_205 = rt0_0_205 * (2 + (input[i] > (822+offset0)));
t0_0_205 += (1 - rt0_0_205) * (input[i] > (820+offset0));

int rt0_0_206 = (input[i] > (825+offset0));
int t0_0_206 = rt0_0_206 * (2 + (input[i] > (826+offset0)));
t0_0_206 += (1 - rt0_0_206) * (input[i] > (824+offset0));

int rt0_0_207 = (input[i] > (829+offset0));
int t0_0_207 = rt0_0_207 * (2 + (input[i] > (830+offset0)));
t0_0_207 += (1 - rt0_0_207) * (input[i] > (828+offset0));

int rt0_0_208 = (input[i] > (833+offset0));
int t0_0_208 = rt0_0_208 * (2 + (input[i] > (834+offset0)));
t0_0_208 += (1 - rt0_0_208) * (input[i] > (832+offset0));

int rt0_0_209 = (input[i] > (837+offset0));
int t0_0_209 = rt0_0_209 * (2 + (input[i] > (838+offset0)));
t0_0_209 += (1 - rt0_0_209) * (input[i] > (836+offset0));

int rt0_0_210 = (input[i] > (841+offset0));
int t0_0_210 = rt0_0_210 * (2 + (input[i] > (842+offset0)));
t0_0_210 += (1 - rt0_0_210) * (input[i] > (840+offset0));

int rt0_0_211 = (input[i] > (845+offset0));
int t0_0_211 = rt0_0_211 * (2 + (input[i] > (846+offset0)));
t0_0_211 += (1 - rt0_0_211) * (input[i] > (844+offset0));

int rt0_0_212 = (input[i] > (849+offset0));
int t0_0_212 = rt0_0_212 * (2 + (input[i] > (850+offset0)));
t0_0_212 += (1 - rt0_0_212) * (input[i] > (848+offset0));

int rt0_0_213 = (input[i] > (853+offset0));
int t0_0_213 = rt0_0_213 * (2 + (input[i] > (854+offset0)));
t0_0_213 += (1 - rt0_0_213) * (input[i] > (852+offset0));

int rt0_0_214 = (input[i] > (857+offset0));
int t0_0_214 = rt0_0_214 * (2 + (input[i] > (858+offset0)));
t0_0_214 += (1 - rt0_0_214) * (input[i] > (856+offset0));

int rt0_0_215 = (input[i] > (861+offset0));
int t0_0_215 = rt0_0_215 * (2 + (input[i] > (862+offset0)));
t0_0_215 += (1 - rt0_0_215) * (input[i] > (860+offset0));

int rt0_0_216 = (input[i] > (865+offset0));
int t0_0_216 = rt0_0_216 * (2 + (input[i] > (866+offset0)));
t0_0_216 += (1 - rt0_0_216) * (input[i] > (864+offset0));

int rt0_0_217 = (input[i] > (869+offset0));
int t0_0_217 = rt0_0_217 * (2 + (input[i] > (870+offset0)));
t0_0_217 += (1 - rt0_0_217) * (input[i] > (868+offset0));

int rt0_0_218 = (input[i] > (873+offset0));
int t0_0_218 = rt0_0_218 * (2 + (input[i] > (874+offset0)));
t0_0_218 += (1 - rt0_0_218) * (input[i] > (872+offset0));

int rt0_0_219 = (input[i] > (877+offset0));
int t0_0_219 = rt0_0_219 * (2 + (input[i] > (878+offset0)));
t0_0_219 += (1 - rt0_0_219) * (input[i] > (876+offset0));

int rt0_0_220 = (input[i] > (881+offset0));
int t0_0_220 = rt0_0_220 * (2 + (input[i] > (882+offset0)));
t0_0_220 += (1 - rt0_0_220) * (input[i] > (880+offset0));

int rt0_0_221 = (input[i] > (885+offset0));
int t0_0_221 = rt0_0_221 * (2 + (input[i] > (886+offset0)));
t0_0_221 += (1 - rt0_0_221) * (input[i] > (884+offset0));

int rt0_0_222 = (input[i] > (889+offset0));
int t0_0_222 = rt0_0_222 * (2 + (input[i] > (890+offset0)));
t0_0_222 += (1 - rt0_0_222) * (input[i] > (888+offset0));

int rt0_0_223 = (input[i] > (893+offset0));
int t0_0_223 = rt0_0_223 * (2 + (input[i] > (894+offset0)));
t0_0_223 += (1 - rt0_0_223) * (input[i] > (892+offset0));

int rt0_0_224 = (input[i] > (897+offset0));
int t0_0_224 = rt0_0_224 * (2 + (input[i] > (898+offset0)));
t0_0_224 += (1 - rt0_0_224) * (input[i] > (896+offset0));

int rt0_0_225 = (input[i] > (901+offset0));
int t0_0_225 = rt0_0_225 * (2 + (input[i] > (902+offset0)));
t0_0_225 += (1 - rt0_0_225) * (input[i] > (900+offset0));

int rt0_0_226 = (input[i] > (905+offset0));
int t0_0_226 = rt0_0_226 * (2 + (input[i] > (906+offset0)));
t0_0_226 += (1 - rt0_0_226) * (input[i] > (904+offset0));

int rt0_0_227 = (input[i] > (909+offset0));
int t0_0_227 = rt0_0_227 * (2 + (input[i] > (910+offset0)));
t0_0_227 += (1 - rt0_0_227) * (input[i] > (908+offset0));

int rt0_0_228 = (input[i] > (913+offset0));
int t0_0_228 = rt0_0_228 * (2 + (input[i] > (914+offset0)));
t0_0_228 += (1 - rt0_0_228) * (input[i] > (912+offset0));

int rt0_0_229 = (input[i] > (917+offset0));
int t0_0_229 = rt0_0_229 * (2 + (input[i] > (918+offset0)));
t0_0_229 += (1 - rt0_0_229) * (input[i] > (916+offset0));

int rt0_0_230 = (input[i] > (921+offset0));
int t0_0_230 = rt0_0_230 * (2 + (input[i] > (922+offset0)));
t0_0_230 += (1 - rt0_0_230) * (input[i] > (920+offset0));

int rt0_0_231 = (input[i] > (925+offset0));
int t0_0_231 = rt0_0_231 * (2 + (input[i] > (926+offset0)));
t0_0_231 += (1 - rt0_0_231) * (input[i] > (924+offset0));

int rt0_0_232 = (input[i] > (929+offset0));
int t0_0_232 = rt0_0_232 * (2 + (input[i] > (930+offset0)));
t0_0_232 += (1 - rt0_0_232) * (input[i] > (928+offset0));

int rt0_0_233 = (input[i] > (933+offset0));
int t0_0_233 = rt0_0_233 * (2 + (input[i] > (934+offset0)));
t0_0_233 += (1 - rt0_0_233) * (input[i] > (932+offset0));

int rt0_0_234 = (input[i] > (937+offset0));
int t0_0_234 = rt0_0_234 * (2 + (input[i] > (938+offset0)));
t0_0_234 += (1 - rt0_0_234) * (input[i] > (936+offset0));

int rt0_0_235 = (input[i] > (941+offset0));
int t0_0_235 = rt0_0_235 * (2 + (input[i] > (942+offset0)));
t0_0_235 += (1 - rt0_0_235) * (input[i] > (940+offset0));

int rt0_0_236 = (input[i] > (945+offset0));
int t0_0_236 = rt0_0_236 * (2 + (input[i] > (946+offset0)));
t0_0_236 += (1 - rt0_0_236) * (input[i] > (944+offset0));

int rt0_0_237 = (input[i] > (949+offset0));
int t0_0_237 = rt0_0_237 * (2 + (input[i] > (950+offset0)));
t0_0_237 += (1 - rt0_0_237) * (input[i] > (948+offset0));

int rt0_0_238 = (input[i] > (953+offset0));
int t0_0_238 = rt0_0_238 * (2 + (input[i] > (954+offset0)));
t0_0_238 += (1 - rt0_0_238) * (input[i] > (952+offset0));

int rt0_0_239 = (input[i] > (957+offset0));
int t0_0_239 = rt0_0_239 * (2 + (input[i] > (958+offset0)));
t0_0_239 += (1 - rt0_0_239) * (input[i] > (956+offset0));

int rt0_0_240 = (input[i] > (961+offset0));
int t0_0_240 = rt0_0_240 * (2 + (input[i] > (962+offset0)));
t0_0_240 += (1 - rt0_0_240) * (input[i] > (960+offset0));

int rt0_0_241 = (input[i] > (965+offset0));
int t0_0_241 = rt0_0_241 * (2 + (input[i] > (966+offset0)));
t0_0_241 += (1 - rt0_0_241) * (input[i] > (964+offset0));

int rt0_0_242 = (input[i] > (969+offset0));
int t0_0_242 = rt0_0_242 * (2 + (input[i] > (970+offset0)));
t0_0_242 += (1 - rt0_0_242) * (input[i] > (968+offset0));

int rt0_0_243 = (input[i] > (973+offset0));
int t0_0_243 = rt0_0_243 * (2 + (input[i] > (974+offset0)));
t0_0_243 += (1 - rt0_0_243) * (input[i] > (972+offset0));

int rt0_0_244 = (input[i] > (977+offset0));
int t0_0_244 = rt0_0_244 * (2 + (input[i] > (978+offset0)));
t0_0_244 += (1 - rt0_0_244) * (input[i] > (976+offset0));

int rt0_0_245 = (input[i] > (981+offset0));
int t0_0_245 = rt0_0_245 * (2 + (input[i] > (982+offset0)));
t0_0_245 += (1 - rt0_0_245) * (input[i] > (980+offset0));

int rt0_0_246 = (input[i] > (985+offset0));
int t0_0_246 = rt0_0_246 * (2 + (input[i] > (986+offset0)));
t0_0_246 += (1 - rt0_0_246) * (input[i] > (984+offset0));

int rt0_0_247 = (input[i] > (989+offset0));
int t0_0_247 = rt0_0_247 * (2 + (input[i] > (990+offset0)));
t0_0_247 += (1 - rt0_0_247) * (input[i] > (988+offset0));

int rt0_0_248 = (input[i] > (993+offset0));
int t0_0_248 = rt0_0_248 * (2 + (input[i] > (994+offset0)));
t0_0_248 += (1 - rt0_0_248) * (input[i] > (992+offset0));

int rt0_0_249 = (input[i] > (997+offset0));
int t0_0_249 = rt0_0_249 * (2 + (input[i] > (998+offset0)));
t0_0_249 += (1 - rt0_0_249) * (input[i] > (996+offset0));

int rt0_0_250 = (input[i] > (1001+offset0));
int t0_0_250 = rt0_0_250 * (2 + (input[i] > (1002+offset0)));
t0_0_250 += (1 - rt0_0_250) * (input[i] > (1000+offset0));

int rt0_0_251 = (input[i] > (1005+offset0));
int t0_0_251 = rt0_0_251 * (2 + (input[i] > (1006+offset0)));
t0_0_251 += (1 - rt0_0_251) * (input[i] > (1004+offset0));

int rt0_0_252 = (input[i] > (1009+offset0));
int t0_0_252 = rt0_0_252 * (2 + (input[i] > (1010+offset0)));
t0_0_252 += (1 - rt0_0_252) * (input[i] > (1008+offset0));

int rt0_0_253 = (input[i] > (1013+offset0));
int t0_0_253 = rt0_0_253 * (2 + (input[i] > (1014+offset0)));
t0_0_253 += (1 - rt0_0_253) * (input[i] > (1012+offset0));

int rt0_0_254 = (input[i] > (1017+offset0));
int t0_0_254 = rt0_0_254 * (2 + (input[i] > (1018+offset0)));
t0_0_254 += (1 - rt0_0_254) * (input[i] > (1016+offset0));

int rt0_0_255 = (input[i] > (1021+offset0));
int t0_0_255 = rt0_0_255 * (2 + (input[i] > (1022+offset0)));
t0_0_255 += (1 - rt0_0_255) * (input[i] > (1020+offset0));

int rt0_1_0 = (input[i] > (3+offset0));
int t0_1_0 = rt0_1_0 * (4 + t0_0_1);
t0_1_0 += (1 - rt0_1_0) * t0_0_0;

int rt0_1_1 = (input[i] > (11+offset0));
int t0_1_1 = rt0_1_1 * (4 + t0_0_3);
t0_1_1 += (1 - rt0_1_1) * t0_0_2;

int rt0_1_2 = (input[i] > (19+offset0));
int t0_1_2 = rt0_1_2 * (4 + t0_0_5);
t0_1_2 += (1 - rt0_1_2) * t0_0_4;

int rt0_1_3 = (input[i] > (27+offset0));
int t0_1_3 = rt0_1_3 * (4 + t0_0_7);
t0_1_3 += (1 - rt0_1_3) * t0_0_6;

int rt0_1_4 = (input[i] > (35+offset0));
int t0_1_4 = rt0_1_4 * (4 + t0_0_9);
t0_1_4 += (1 - rt0_1_4) * t0_0_8;

int rt0_1_5 = (input[i] > (43+offset0));
int t0_1_5 = rt0_1_5 * (4 + t0_0_11);
t0_1_5 += (1 - rt0_1_5) * t0_0_10;

int rt0_1_6 = (input[i] > (51+offset0));
int t0_1_6 = rt0_1_6 * (4 + t0_0_13);
t0_1_6 += (1 - rt0_1_6) * t0_0_12;

int rt0_1_7 = (input[i] > (59+offset0));
int t0_1_7 = rt0_1_7 * (4 + t0_0_15);
t0_1_7 += (1 - rt0_1_7) * t0_0_14;

int rt0_1_8 = (input[i] > (67+offset0));
int t0_1_8 = rt0_1_8 * (4 + t0_0_17);
t0_1_8 += (1 - rt0_1_8) * t0_0_16;

int rt0_1_9 = (input[i] > (75+offset0));
int t0_1_9 = rt0_1_9 * (4 + t0_0_19);
t0_1_9 += (1 - rt0_1_9) * t0_0_18;

int rt0_1_10 = (input[i] > (83+offset0));
int t0_1_10 = rt0_1_10 * (4 + t0_0_21);
t0_1_10 += (1 - rt0_1_10) * t0_0_20;

int rt0_1_11 = (input[i] > (91+offset0));
int t0_1_11 = rt0_1_11 * (4 + t0_0_23);
t0_1_11 += (1 - rt0_1_11) * t0_0_22;

int rt0_1_12 = (input[i] > (99+offset0));
int t0_1_12 = rt0_1_12 * (4 + t0_0_25);
t0_1_12 += (1 - rt0_1_12) * t0_0_24;

int rt0_1_13 = (input[i] > (107+offset0));
int t0_1_13 = rt0_1_13 * (4 + t0_0_27);
t0_1_13 += (1 - rt0_1_13) * t0_0_26;

int rt0_1_14 = (input[i] > (115+offset0));
int t0_1_14 = rt0_1_14 * (4 + t0_0_29);
t0_1_14 += (1 - rt0_1_14) * t0_0_28;

int rt0_1_15 = (input[i] > (123+offset0));
int t0_1_15 = rt0_1_15 * (4 + t0_0_31);
t0_1_15 += (1 - rt0_1_15) * t0_0_30;

int rt0_1_16 = (input[i] > (131+offset0));
int t0_1_16 = rt0_1_16 * (4 + t0_0_33);
t0_1_16 += (1 - rt0_1_16) * t0_0_32;

int rt0_1_17 = (input[i] > (139+offset0));
int t0_1_17 = rt0_1_17 * (4 + t0_0_35);
t0_1_17 += (1 - rt0_1_17) * t0_0_34;

int rt0_1_18 = (input[i] > (147+offset0));
int t0_1_18 = rt0_1_18 * (4 + t0_0_37);
t0_1_18 += (1 - rt0_1_18) * t0_0_36;

int rt0_1_19 = (input[i] > (155+offset0));
int t0_1_19 = rt0_1_19 * (4 + t0_0_39);
t0_1_19 += (1 - rt0_1_19) * t0_0_38;

int rt0_1_20 = (input[i] > (163+offset0));
int t0_1_20 = rt0_1_20 * (4 + t0_0_41);
t0_1_20 += (1 - rt0_1_20) * t0_0_40;

int rt0_1_21 = (input[i] > (171+offset0));
int t0_1_21 = rt0_1_21 * (4 + t0_0_43);
t0_1_21 += (1 - rt0_1_21) * t0_0_42;

int rt0_1_22 = (input[i] > (179+offset0));
int t0_1_22 = rt0_1_22 * (4 + t0_0_45);
t0_1_22 += (1 - rt0_1_22) * t0_0_44;

int rt0_1_23 = (input[i] > (187+offset0));
int t0_1_23 = rt0_1_23 * (4 + t0_0_47);
t0_1_23 += (1 - rt0_1_23) * t0_0_46;

int rt0_1_24 = (input[i] > (195+offset0));
int t0_1_24 = rt0_1_24 * (4 + t0_0_49);
t0_1_24 += (1 - rt0_1_24) * t0_0_48;

int rt0_1_25 = (input[i] > (203+offset0));
int t0_1_25 = rt0_1_25 * (4 + t0_0_51);
t0_1_25 += (1 - rt0_1_25) * t0_0_50;

int rt0_1_26 = (input[i] > (211+offset0));
int t0_1_26 = rt0_1_26 * (4 + t0_0_53);
t0_1_26 += (1 - rt0_1_26) * t0_0_52;

int rt0_1_27 = (input[i] > (219+offset0));
int t0_1_27 = rt0_1_27 * (4 + t0_0_55);
t0_1_27 += (1 - rt0_1_27) * t0_0_54;

int rt0_1_28 = (input[i] > (227+offset0));
int t0_1_28 = rt0_1_28 * (4 + t0_0_57);
t0_1_28 += (1 - rt0_1_28) * t0_0_56;

int rt0_1_29 = (input[i] > (235+offset0));
int t0_1_29 = rt0_1_29 * (4 + t0_0_59);
t0_1_29 += (1 - rt0_1_29) * t0_0_58;

int rt0_1_30 = (input[i] > (243+offset0));
int t0_1_30 = rt0_1_30 * (4 + t0_0_61);
t0_1_30 += (1 - rt0_1_30) * t0_0_60;

int rt0_1_31 = (input[i] > (251+offset0));
int t0_1_31 = rt0_1_31 * (4 + t0_0_63);
t0_1_31 += (1 - rt0_1_31) * t0_0_62;

int rt0_1_32 = (input[i] > (259+offset0));
int t0_1_32 = rt0_1_32 * (4 + t0_0_65);
t0_1_32 += (1 - rt0_1_32) * t0_0_64;

int rt0_1_33 = (input[i] > (267+offset0));
int t0_1_33 = rt0_1_33 * (4 + t0_0_67);
t0_1_33 += (1 - rt0_1_33) * t0_0_66;

int rt0_1_34 = (input[i] > (275+offset0));
int t0_1_34 = rt0_1_34 * (4 + t0_0_69);
t0_1_34 += (1 - rt0_1_34) * t0_0_68;

int rt0_1_35 = (input[i] > (283+offset0));
int t0_1_35 = rt0_1_35 * (4 + t0_0_71);
t0_1_35 += (1 - rt0_1_35) * t0_0_70;

int rt0_1_36 = (input[i] > (291+offset0));
int t0_1_36 = rt0_1_36 * (4 + t0_0_73);
t0_1_36 += (1 - rt0_1_36) * t0_0_72;

int rt0_1_37 = (input[i] > (299+offset0));
int t0_1_37 = rt0_1_37 * (4 + t0_0_75);
t0_1_37 += (1 - rt0_1_37) * t0_0_74;

int rt0_1_38 = (input[i] > (307+offset0));
int t0_1_38 = rt0_1_38 * (4 + t0_0_77);
t0_1_38 += (1 - rt0_1_38) * t0_0_76;

int rt0_1_39 = (input[i] > (315+offset0));
int t0_1_39 = rt0_1_39 * (4 + t0_0_79);
t0_1_39 += (1 - rt0_1_39) * t0_0_78;

int rt0_1_40 = (input[i] > (323+offset0));
int t0_1_40 = rt0_1_40 * (4 + t0_0_81);
t0_1_40 += (1 - rt0_1_40) * t0_0_80;

int rt0_1_41 = (input[i] > (331+offset0));
int t0_1_41 = rt0_1_41 * (4 + t0_0_83);
t0_1_41 += (1 - rt0_1_41) * t0_0_82;

int rt0_1_42 = (input[i] > (339+offset0));
int t0_1_42 = rt0_1_42 * (4 + t0_0_85);
t0_1_42 += (1 - rt0_1_42) * t0_0_84;

int rt0_1_43 = (input[i] > (347+offset0));
int t0_1_43 = rt0_1_43 * (4 + t0_0_87);
t0_1_43 += (1 - rt0_1_43) * t0_0_86;

int rt0_1_44 = (input[i] > (355+offset0));
int t0_1_44 = rt0_1_44 * (4 + t0_0_89);
t0_1_44 += (1 - rt0_1_44) * t0_0_88;

int rt0_1_45 = (input[i] > (363+offset0));
int t0_1_45 = rt0_1_45 * (4 + t0_0_91);
t0_1_45 += (1 - rt0_1_45) * t0_0_90;

int rt0_1_46 = (input[i] > (371+offset0));
int t0_1_46 = rt0_1_46 * (4 + t0_0_93);
t0_1_46 += (1 - rt0_1_46) * t0_0_92;

int rt0_1_47 = (input[i] > (379+offset0));
int t0_1_47 = rt0_1_47 * (4 + t0_0_95);
t0_1_47 += (1 - rt0_1_47) * t0_0_94;

int rt0_1_48 = (input[i] > (387+offset0));
int t0_1_48 = rt0_1_48 * (4 + t0_0_97);
t0_1_48 += (1 - rt0_1_48) * t0_0_96;

int rt0_1_49 = (input[i] > (395+offset0));
int t0_1_49 = rt0_1_49 * (4 + t0_0_99);
t0_1_49 += (1 - rt0_1_49) * t0_0_98;

int rt0_1_50 = (input[i] > (403+offset0));
int t0_1_50 = rt0_1_50 * (4 + t0_0_101);
t0_1_50 += (1 - rt0_1_50) * t0_0_100;

int rt0_1_51 = (input[i] > (411+offset0));
int t0_1_51 = rt0_1_51 * (4 + t0_0_103);
t0_1_51 += (1 - rt0_1_51) * t0_0_102;

int rt0_1_52 = (input[i] > (419+offset0));
int t0_1_52 = rt0_1_52 * (4 + t0_0_105);
t0_1_52 += (1 - rt0_1_52) * t0_0_104;

int rt0_1_53 = (input[i] > (427+offset0));
int t0_1_53 = rt0_1_53 * (4 + t0_0_107);
t0_1_53 += (1 - rt0_1_53) * t0_0_106;

int rt0_1_54 = (input[i] > (435+offset0));
int t0_1_54 = rt0_1_54 * (4 + t0_0_109);
t0_1_54 += (1 - rt0_1_54) * t0_0_108;

int rt0_1_55 = (input[i] > (443+offset0));
int t0_1_55 = rt0_1_55 * (4 + t0_0_111);
t0_1_55 += (1 - rt0_1_55) * t0_0_110;

int rt0_1_56 = (input[i] > (451+offset0));
int t0_1_56 = rt0_1_56 * (4 + t0_0_113);
t0_1_56 += (1 - rt0_1_56) * t0_0_112;

int rt0_1_57 = (input[i] > (459+offset0));
int t0_1_57 = rt0_1_57 * (4 + t0_0_115);
t0_1_57 += (1 - rt0_1_57) * t0_0_114;

int rt0_1_58 = (input[i] > (467+offset0));
int t0_1_58 = rt0_1_58 * (4 + t0_0_117);
t0_1_58 += (1 - rt0_1_58) * t0_0_116;

int rt0_1_59 = (input[i] > (475+offset0));
int t0_1_59 = rt0_1_59 * (4 + t0_0_119);
t0_1_59 += (1 - rt0_1_59) * t0_0_118;

int rt0_1_60 = (input[i] > (483+offset0));
int t0_1_60 = rt0_1_60 * (4 + t0_0_121);
t0_1_60 += (1 - rt0_1_60) * t0_0_120;

int rt0_1_61 = (input[i] > (491+offset0));
int t0_1_61 = rt0_1_61 * (4 + t0_0_123);
t0_1_61 += (1 - rt0_1_61) * t0_0_122;

int rt0_1_62 = (input[i] > (499+offset0));
int t0_1_62 = rt0_1_62 * (4 + t0_0_125);
t0_1_62 += (1 - rt0_1_62) * t0_0_124;

int rt0_1_63 = (input[i] > (507+offset0));
int t0_1_63 = rt0_1_63 * (4 + t0_0_127);
t0_1_63 += (1 - rt0_1_63) * t0_0_126;

int rt0_1_64 = (input[i] > (515+offset0));
int t0_1_64 = rt0_1_64 * (4 + t0_0_129);
t0_1_64 += (1 - rt0_1_64) * t0_0_128;

int rt0_1_65 = (input[i] > (523+offset0));
int t0_1_65 = rt0_1_65 * (4 + t0_0_131);
t0_1_65 += (1 - rt0_1_65) * t0_0_130;

int rt0_1_66 = (input[i] > (531+offset0));
int t0_1_66 = rt0_1_66 * (4 + t0_0_133);
t0_1_66 += (1 - rt0_1_66) * t0_0_132;

int rt0_1_67 = (input[i] > (539+offset0));
int t0_1_67 = rt0_1_67 * (4 + t0_0_135);
t0_1_67 += (1 - rt0_1_67) * t0_0_134;

int rt0_1_68 = (input[i] > (547+offset0));
int t0_1_68 = rt0_1_68 * (4 + t0_0_137);
t0_1_68 += (1 - rt0_1_68) * t0_0_136;

int rt0_1_69 = (input[i] > (555+offset0));
int t0_1_69 = rt0_1_69 * (4 + t0_0_139);
t0_1_69 += (1 - rt0_1_69) * t0_0_138;

int rt0_1_70 = (input[i] > (563+offset0));
int t0_1_70 = rt0_1_70 * (4 + t0_0_141);
t0_1_70 += (1 - rt0_1_70) * t0_0_140;

int rt0_1_71 = (input[i] > (571+offset0));
int t0_1_71 = rt0_1_71 * (4 + t0_0_143);
t0_1_71 += (1 - rt0_1_71) * t0_0_142;

int rt0_1_72 = (input[i] > (579+offset0));
int t0_1_72 = rt0_1_72 * (4 + t0_0_145);
t0_1_72 += (1 - rt0_1_72) * t0_0_144;

int rt0_1_73 = (input[i] > (587+offset0));
int t0_1_73 = rt0_1_73 * (4 + t0_0_147);
t0_1_73 += (1 - rt0_1_73) * t0_0_146;

int rt0_1_74 = (input[i] > (595+offset0));
int t0_1_74 = rt0_1_74 * (4 + t0_0_149);
t0_1_74 += (1 - rt0_1_74) * t0_0_148;

int rt0_1_75 = (input[i] > (603+offset0));
int t0_1_75 = rt0_1_75 * (4 + t0_0_151);
t0_1_75 += (1 - rt0_1_75) * t0_0_150;

int rt0_1_76 = (input[i] > (611+offset0));
int t0_1_76 = rt0_1_76 * (4 + t0_0_153);
t0_1_76 += (1 - rt0_1_76) * t0_0_152;

int rt0_1_77 = (input[i] > (619+offset0));
int t0_1_77 = rt0_1_77 * (4 + t0_0_155);
t0_1_77 += (1 - rt0_1_77) * t0_0_154;

int rt0_1_78 = (input[i] > (627+offset0));
int t0_1_78 = rt0_1_78 * (4 + t0_0_157);
t0_1_78 += (1 - rt0_1_78) * t0_0_156;

int rt0_1_79 = (input[i] > (635+offset0));
int t0_1_79 = rt0_1_79 * (4 + t0_0_159);
t0_1_79 += (1 - rt0_1_79) * t0_0_158;

int rt0_1_80 = (input[i] > (643+offset0));
int t0_1_80 = rt0_1_80 * (4 + t0_0_161);
t0_1_80 += (1 - rt0_1_80) * t0_0_160;

int rt0_1_81 = (input[i] > (651+offset0));
int t0_1_81 = rt0_1_81 * (4 + t0_0_163);
t0_1_81 += (1 - rt0_1_81) * t0_0_162;

int rt0_1_82 = (input[i] > (659+offset0));
int t0_1_82 = rt0_1_82 * (4 + t0_0_165);
t0_1_82 += (1 - rt0_1_82) * t0_0_164;

int rt0_1_83 = (input[i] > (667+offset0));
int t0_1_83 = rt0_1_83 * (4 + t0_0_167);
t0_1_83 += (1 - rt0_1_83) * t0_0_166;

int rt0_1_84 = (input[i] > (675+offset0));
int t0_1_84 = rt0_1_84 * (4 + t0_0_169);
t0_1_84 += (1 - rt0_1_84) * t0_0_168;

int rt0_1_85 = (input[i] > (683+offset0));
int t0_1_85 = rt0_1_85 * (4 + t0_0_171);
t0_1_85 += (1 - rt0_1_85) * t0_0_170;

int rt0_1_86 = (input[i] > (691+offset0));
int t0_1_86 = rt0_1_86 * (4 + t0_0_173);
t0_1_86 += (1 - rt0_1_86) * t0_0_172;

int rt0_1_87 = (input[i] > (699+offset0));
int t0_1_87 = rt0_1_87 * (4 + t0_0_175);
t0_1_87 += (1 - rt0_1_87) * t0_0_174;

int rt0_1_88 = (input[i] > (707+offset0));
int t0_1_88 = rt0_1_88 * (4 + t0_0_177);
t0_1_88 += (1 - rt0_1_88) * t0_0_176;

int rt0_1_89 = (input[i] > (715+offset0));
int t0_1_89 = rt0_1_89 * (4 + t0_0_179);
t0_1_89 += (1 - rt0_1_89) * t0_0_178;

int rt0_1_90 = (input[i] > (723+offset0));
int t0_1_90 = rt0_1_90 * (4 + t0_0_181);
t0_1_90 += (1 - rt0_1_90) * t0_0_180;

int rt0_1_91 = (input[i] > (731+offset0));
int t0_1_91 = rt0_1_91 * (4 + t0_0_183);
t0_1_91 += (1 - rt0_1_91) * t0_0_182;

int rt0_1_92 = (input[i] > (739+offset0));
int t0_1_92 = rt0_1_92 * (4 + t0_0_185);
t0_1_92 += (1 - rt0_1_92) * t0_0_184;

int rt0_1_93 = (input[i] > (747+offset0));
int t0_1_93 = rt0_1_93 * (4 + t0_0_187);
t0_1_93 += (1 - rt0_1_93) * t0_0_186;

int rt0_1_94 = (input[i] > (755+offset0));
int t0_1_94 = rt0_1_94 * (4 + t0_0_189);
t0_1_94 += (1 - rt0_1_94) * t0_0_188;

int rt0_1_95 = (input[i] > (763+offset0));
int t0_1_95 = rt0_1_95 * (4 + t0_0_191);
t0_1_95 += (1 - rt0_1_95) * t0_0_190;

int rt0_1_96 = (input[i] > (771+offset0));
int t0_1_96 = rt0_1_96 * (4 + t0_0_193);
t0_1_96 += (1 - rt0_1_96) * t0_0_192;

int rt0_1_97 = (input[i] > (779+offset0));
int t0_1_97 = rt0_1_97 * (4 + t0_0_195);
t0_1_97 += (1 - rt0_1_97) * t0_0_194;

int rt0_1_98 = (input[i] > (787+offset0));
int t0_1_98 = rt0_1_98 * (4 + t0_0_197);
t0_1_98 += (1 - rt0_1_98) * t0_0_196;

int rt0_1_99 = (input[i] > (795+offset0));
int t0_1_99 = rt0_1_99 * (4 + t0_0_199);
t0_1_99 += (1 - rt0_1_99) * t0_0_198;

int rt0_1_100 = (input[i] > (803+offset0));
int t0_1_100 = rt0_1_100 * (4 + t0_0_201);
t0_1_100 += (1 - rt0_1_100) * t0_0_200;

int rt0_1_101 = (input[i] > (811+offset0));
int t0_1_101 = rt0_1_101 * (4 + t0_0_203);
t0_1_101 += (1 - rt0_1_101) * t0_0_202;

int rt0_1_102 = (input[i] > (819+offset0));
int t0_1_102 = rt0_1_102 * (4 + t0_0_205);
t0_1_102 += (1 - rt0_1_102) * t0_0_204;

int rt0_1_103 = (input[i] > (827+offset0));
int t0_1_103 = rt0_1_103 * (4 + t0_0_207);
t0_1_103 += (1 - rt0_1_103) * t0_0_206;

int rt0_1_104 = (input[i] > (835+offset0));
int t0_1_104 = rt0_1_104 * (4 + t0_0_209);
t0_1_104 += (1 - rt0_1_104) * t0_0_208;

int rt0_1_105 = (input[i] > (843+offset0));
int t0_1_105 = rt0_1_105 * (4 + t0_0_211);
t0_1_105 += (1 - rt0_1_105) * t0_0_210;

int rt0_1_106 = (input[i] > (851+offset0));
int t0_1_106 = rt0_1_106 * (4 + t0_0_213);
t0_1_106 += (1 - rt0_1_106) * t0_0_212;

int rt0_1_107 = (input[i] > (859+offset0));
int t0_1_107 = rt0_1_107 * (4 + t0_0_215);
t0_1_107 += (1 - rt0_1_107) * t0_0_214;

int rt0_1_108 = (input[i] > (867+offset0));
int t0_1_108 = rt0_1_108 * (4 + t0_0_217);
t0_1_108 += (1 - rt0_1_108) * t0_0_216;

int rt0_1_109 = (input[i] > (875+offset0));
int t0_1_109 = rt0_1_109 * (4 + t0_0_219);
t0_1_109 += (1 - rt0_1_109) * t0_0_218;

int rt0_1_110 = (input[i] > (883+offset0));
int t0_1_110 = rt0_1_110 * (4 + t0_0_221);
t0_1_110 += (1 - rt0_1_110) * t0_0_220;

int rt0_1_111 = (input[i] > (891+offset0));
int t0_1_111 = rt0_1_111 * (4 + t0_0_223);
t0_1_111 += (1 - rt0_1_111) * t0_0_222;

int rt0_1_112 = (input[i] > (899+offset0));
int t0_1_112 = rt0_1_112 * (4 + t0_0_225);
t0_1_112 += (1 - rt0_1_112) * t0_0_224;

int rt0_1_113 = (input[i] > (907+offset0));
int t0_1_113 = rt0_1_113 * (4 + t0_0_227);
t0_1_113 += (1 - rt0_1_113) * t0_0_226;

int rt0_1_114 = (input[i] > (915+offset0));
int t0_1_114 = rt0_1_114 * (4 + t0_0_229);
t0_1_114 += (1 - rt0_1_114) * t0_0_228;

int rt0_1_115 = (input[i] > (923+offset0));
int t0_1_115 = rt0_1_115 * (4 + t0_0_231);
t0_1_115 += (1 - rt0_1_115) * t0_0_230;

int rt0_1_116 = (input[i] > (931+offset0));
int t0_1_116 = rt0_1_116 * (4 + t0_0_233);
t0_1_116 += (1 - rt0_1_116) * t0_0_232;

int rt0_1_117 = (input[i] > (939+offset0));
int t0_1_117 = rt0_1_117 * (4 + t0_0_235);
t0_1_117 += (1 - rt0_1_117) * t0_0_234;

int rt0_1_118 = (input[i] > (947+offset0));
int t0_1_118 = rt0_1_118 * (4 + t0_0_237);
t0_1_118 += (1 - rt0_1_118) * t0_0_236;

int rt0_1_119 = (input[i] > (955+offset0));
int t0_1_119 = rt0_1_119 * (4 + t0_0_239);
t0_1_119 += (1 - rt0_1_119) * t0_0_238;

int rt0_1_120 = (input[i] > (963+offset0));
int t0_1_120 = rt0_1_120 * (4 + t0_0_241);
t0_1_120 += (1 - rt0_1_120) * t0_0_240;

int rt0_1_121 = (input[i] > (971+offset0));
int t0_1_121 = rt0_1_121 * (4 + t0_0_243);
t0_1_121 += (1 - rt0_1_121) * t0_0_242;

int rt0_1_122 = (input[i] > (979+offset0));
int t0_1_122 = rt0_1_122 * (4 + t0_0_245);
t0_1_122 += (1 - rt0_1_122) * t0_0_244;

int rt0_1_123 = (input[i] > (987+offset0));
int t0_1_123 = rt0_1_123 * (4 + t0_0_247);
t0_1_123 += (1 - rt0_1_123) * t0_0_246;

int rt0_1_124 = (input[i] > (995+offset0));
int t0_1_124 = rt0_1_124 * (4 + t0_0_249);
t0_1_124 += (1 - rt0_1_124) * t0_0_248;

int rt0_1_125 = (input[i] > (1003+offset0));
int t0_1_125 = rt0_1_125 * (4 + t0_0_251);
t0_1_125 += (1 - rt0_1_125) * t0_0_250;

int rt0_1_126 = (input[i] > (1011+offset0));
int t0_1_126 = rt0_1_126 * (4 + t0_0_253);
t0_1_126 += (1 - rt0_1_126) * t0_0_252;

int rt0_1_127 = (input[i] > (1019+offset0));
int t0_1_127 = rt0_1_127 * (4 + t0_0_255);
t0_1_127 += (1 - rt0_1_127) * t0_0_254;

int rt0_2_0 = (input[i] > (7+offset0));
int t0_2_0 = rt0_2_0 * (8 + t0_1_1);
t0_2_0 += (1 - rt0_2_0) * t0_1_0;

int rt0_2_1 = (input[i] > (23+offset0));
int t0_2_1 = rt0_2_1 * (8 + t0_1_3);
t0_2_1 += (1 - rt0_2_1) * t0_1_2;

int rt0_2_2 = (input[i] > (39+offset0));
int t0_2_2 = rt0_2_2 * (8 + t0_1_5);
t0_2_2 += (1 - rt0_2_2) * t0_1_4;

int rt0_2_3 = (input[i] > (55+offset0));
int t0_2_3 = rt0_2_3 * (8 + t0_1_7);
t0_2_3 += (1 - rt0_2_3) * t0_1_6;

int rt0_2_4 = (input[i] > (71+offset0));
int t0_2_4 = rt0_2_4 * (8 + t0_1_9);
t0_2_4 += (1 - rt0_2_4) * t0_1_8;

int rt0_2_5 = (input[i] > (87+offset0));
int t0_2_5 = rt0_2_5 * (8 + t0_1_11);
t0_2_5 += (1 - rt0_2_5) * t0_1_10;

int rt0_2_6 = (input[i] > (103+offset0));
int t0_2_6 = rt0_2_6 * (8 + t0_1_13);
t0_2_6 += (1 - rt0_2_6) * t0_1_12;

int rt0_2_7 = (input[i] > (119+offset0));
int t0_2_7 = rt0_2_7 * (8 + t0_1_15);
t0_2_7 += (1 - rt0_2_7) * t0_1_14;

int rt0_2_8 = (input[i] > (135+offset0));
int t0_2_8 = rt0_2_8 * (8 + t0_1_17);
t0_2_8 += (1 - rt0_2_8) * t0_1_16;

int rt0_2_9 = (input[i] > (151+offset0));
int t0_2_9 = rt0_2_9 * (8 + t0_1_19);
t0_2_9 += (1 - rt0_2_9) * t0_1_18;

int rt0_2_10 = (input[i] > (167+offset0));
int t0_2_10 = rt0_2_10 * (8 + t0_1_21);
t0_2_10 += (1 - rt0_2_10) * t0_1_20;

int rt0_2_11 = (input[i] > (183+offset0));
int t0_2_11 = rt0_2_11 * (8 + t0_1_23);
t0_2_11 += (1 - rt0_2_11) * t0_1_22;

int rt0_2_12 = (input[i] > (199+offset0));
int t0_2_12 = rt0_2_12 * (8 + t0_1_25);
t0_2_12 += (1 - rt0_2_12) * t0_1_24;

int rt0_2_13 = (input[i] > (215+offset0));
int t0_2_13 = rt0_2_13 * (8 + t0_1_27);
t0_2_13 += (1 - rt0_2_13) * t0_1_26;

int rt0_2_14 = (input[i] > (231+offset0));
int t0_2_14 = rt0_2_14 * (8 + t0_1_29);
t0_2_14 += (1 - rt0_2_14) * t0_1_28;

int rt0_2_15 = (input[i] > (247+offset0));
int t0_2_15 = rt0_2_15 * (8 + t0_1_31);
t0_2_15 += (1 - rt0_2_15) * t0_1_30;

int rt0_2_16 = (input[i] > (263+offset0));
int t0_2_16 = rt0_2_16 * (8 + t0_1_33);
t0_2_16 += (1 - rt0_2_16) * t0_1_32;

int rt0_2_17 = (input[i] > (279+offset0));
int t0_2_17 = rt0_2_17 * (8 + t0_1_35);
t0_2_17 += (1 - rt0_2_17) * t0_1_34;

int rt0_2_18 = (input[i] > (295+offset0));
int t0_2_18 = rt0_2_18 * (8 + t0_1_37);
t0_2_18 += (1 - rt0_2_18) * t0_1_36;

int rt0_2_19 = (input[i] > (311+offset0));
int t0_2_19 = rt0_2_19 * (8 + t0_1_39);
t0_2_19 += (1 - rt0_2_19) * t0_1_38;

int rt0_2_20 = (input[i] > (327+offset0));
int t0_2_20 = rt0_2_20 * (8 + t0_1_41);
t0_2_20 += (1 - rt0_2_20) * t0_1_40;

int rt0_2_21 = (input[i] > (343+offset0));
int t0_2_21 = rt0_2_21 * (8 + t0_1_43);
t0_2_21 += (1 - rt0_2_21) * t0_1_42;

int rt0_2_22 = (input[i] > (359+offset0));
int t0_2_22 = rt0_2_22 * (8 + t0_1_45);
t0_2_22 += (1 - rt0_2_22) * t0_1_44;

int rt0_2_23 = (input[i] > (375+offset0));
int t0_2_23 = rt0_2_23 * (8 + t0_1_47);
t0_2_23 += (1 - rt0_2_23) * t0_1_46;

int rt0_2_24 = (input[i] > (391+offset0));
int t0_2_24 = rt0_2_24 * (8 + t0_1_49);
t0_2_24 += (1 - rt0_2_24) * t0_1_48;

int rt0_2_25 = (input[i] > (407+offset0));
int t0_2_25 = rt0_2_25 * (8 + t0_1_51);
t0_2_25 += (1 - rt0_2_25) * t0_1_50;

int rt0_2_26 = (input[i] > (423+offset0));
int t0_2_26 = rt0_2_26 * (8 + t0_1_53);
t0_2_26 += (1 - rt0_2_26) * t0_1_52;

int rt0_2_27 = (input[i] > (439+offset0));
int t0_2_27 = rt0_2_27 * (8 + t0_1_55);
t0_2_27 += (1 - rt0_2_27) * t0_1_54;

int rt0_2_28 = (input[i] > (455+offset0));
int t0_2_28 = rt0_2_28 * (8 + t0_1_57);
t0_2_28 += (1 - rt0_2_28) * t0_1_56;

int rt0_2_29 = (input[i] > (471+offset0));
int t0_2_29 = rt0_2_29 * (8 + t0_1_59);
t0_2_29 += (1 - rt0_2_29) * t0_1_58;

int rt0_2_30 = (input[i] > (487+offset0));
int t0_2_30 = rt0_2_30 * (8 + t0_1_61);
t0_2_30 += (1 - rt0_2_30) * t0_1_60;

int rt0_2_31 = (input[i] > (503+offset0));
int t0_2_31 = rt0_2_31 * (8 + t0_1_63);
t0_2_31 += (1 - rt0_2_31) * t0_1_62;

int rt0_2_32 = (input[i] > (519+offset0));
int t0_2_32 = rt0_2_32 * (8 + t0_1_65);
t0_2_32 += (1 - rt0_2_32) * t0_1_64;

int rt0_2_33 = (input[i] > (535+offset0));
int t0_2_33 = rt0_2_33 * (8 + t0_1_67);
t0_2_33 += (1 - rt0_2_33) * t0_1_66;

int rt0_2_34 = (input[i] > (551+offset0));
int t0_2_34 = rt0_2_34 * (8 + t0_1_69);
t0_2_34 += (1 - rt0_2_34) * t0_1_68;

int rt0_2_35 = (input[i] > (567+offset0));
int t0_2_35 = rt0_2_35 * (8 + t0_1_71);
t0_2_35 += (1 - rt0_2_35) * t0_1_70;

int rt0_2_36 = (input[i] > (583+offset0));
int t0_2_36 = rt0_2_36 * (8 + t0_1_73);
t0_2_36 += (1 - rt0_2_36) * t0_1_72;

int rt0_2_37 = (input[i] > (599+offset0));
int t0_2_37 = rt0_2_37 * (8 + t0_1_75);
t0_2_37 += (1 - rt0_2_37) * t0_1_74;

int rt0_2_38 = (input[i] > (615+offset0));
int t0_2_38 = rt0_2_38 * (8 + t0_1_77);
t0_2_38 += (1 - rt0_2_38) * t0_1_76;

int rt0_2_39 = (input[i] > (631+offset0));
int t0_2_39 = rt0_2_39 * (8 + t0_1_79);
t0_2_39 += (1 - rt0_2_39) * t0_1_78;

int rt0_2_40 = (input[i] > (647+offset0));
int t0_2_40 = rt0_2_40 * (8 + t0_1_81);
t0_2_40 += (1 - rt0_2_40) * t0_1_80;

int rt0_2_41 = (input[i] > (663+offset0));
int t0_2_41 = rt0_2_41 * (8 + t0_1_83);
t0_2_41 += (1 - rt0_2_41) * t0_1_82;

int rt0_2_42 = (input[i] > (679+offset0));
int t0_2_42 = rt0_2_42 * (8 + t0_1_85);
t0_2_42 += (1 - rt0_2_42) * t0_1_84;

int rt0_2_43 = (input[i] > (695+offset0));
int t0_2_43 = rt0_2_43 * (8 + t0_1_87);
t0_2_43 += (1 - rt0_2_43) * t0_1_86;

int rt0_2_44 = (input[i] > (711+offset0));
int t0_2_44 = rt0_2_44 * (8 + t0_1_89);
t0_2_44 += (1 - rt0_2_44) * t0_1_88;

int rt0_2_45 = (input[i] > (727+offset0));
int t0_2_45 = rt0_2_45 * (8 + t0_1_91);
t0_2_45 += (1 - rt0_2_45) * t0_1_90;

int rt0_2_46 = (input[i] > (743+offset0));
int t0_2_46 = rt0_2_46 * (8 + t0_1_93);
t0_2_46 += (1 - rt0_2_46) * t0_1_92;

int rt0_2_47 = (input[i] > (759+offset0));
int t0_2_47 = rt0_2_47 * (8 + t0_1_95);
t0_2_47 += (1 - rt0_2_47) * t0_1_94;

int rt0_2_48 = (input[i] > (775+offset0));
int t0_2_48 = rt0_2_48 * (8 + t0_1_97);
t0_2_48 += (1 - rt0_2_48) * t0_1_96;

int rt0_2_49 = (input[i] > (791+offset0));
int t0_2_49 = rt0_2_49 * (8 + t0_1_99);
t0_2_49 += (1 - rt0_2_49) * t0_1_98;

int rt0_2_50 = (input[i] > (807+offset0));
int t0_2_50 = rt0_2_50 * (8 + t0_1_101);
t0_2_50 += (1 - rt0_2_50) * t0_1_100;

int rt0_2_51 = (input[i] > (823+offset0));
int t0_2_51 = rt0_2_51 * (8 + t0_1_103);
t0_2_51 += (1 - rt0_2_51) * t0_1_102;

int rt0_2_52 = (input[i] > (839+offset0));
int t0_2_52 = rt0_2_52 * (8 + t0_1_105);
t0_2_52 += (1 - rt0_2_52) * t0_1_104;

int rt0_2_53 = (input[i] > (855+offset0));
int t0_2_53 = rt0_2_53 * (8 + t0_1_107);
t0_2_53 += (1 - rt0_2_53) * t0_1_106;

int rt0_2_54 = (input[i] > (871+offset0));
int t0_2_54 = rt0_2_54 * (8 + t0_1_109);
t0_2_54 += (1 - rt0_2_54) * t0_1_108;

int rt0_2_55 = (input[i] > (887+offset0));
int t0_2_55 = rt0_2_55 * (8 + t0_1_111);
t0_2_55 += (1 - rt0_2_55) * t0_1_110;

int rt0_2_56 = (input[i] > (903+offset0));
int t0_2_56 = rt0_2_56 * (8 + t0_1_113);
t0_2_56 += (1 - rt0_2_56) * t0_1_112;

int rt0_2_57 = (input[i] > (919+offset0));
int t0_2_57 = rt0_2_57 * (8 + t0_1_115);
t0_2_57 += (1 - rt0_2_57) * t0_1_114;

int rt0_2_58 = (input[i] > (935+offset0));
int t0_2_58 = rt0_2_58 * (8 + t0_1_117);
t0_2_58 += (1 - rt0_2_58) * t0_1_116;

int rt0_2_59 = (input[i] > (951+offset0));
int t0_2_59 = rt0_2_59 * (8 + t0_1_119);
t0_2_59 += (1 - rt0_2_59) * t0_1_118;

int rt0_2_60 = (input[i] > (967+offset0));
int t0_2_60 = rt0_2_60 * (8 + t0_1_121);
t0_2_60 += (1 - rt0_2_60) * t0_1_120;

int rt0_2_61 = (input[i] > (983+offset0));
int t0_2_61 = rt0_2_61 * (8 + t0_1_123);
t0_2_61 += (1 - rt0_2_61) * t0_1_122;

int rt0_2_62 = (input[i] > (999+offset0));
int t0_2_62 = rt0_2_62 * (8 + t0_1_125);
t0_2_62 += (1 - rt0_2_62) * t0_1_124;

int rt0_2_63 = (input[i] > (1015+offset0));
int t0_2_63 = rt0_2_63 * (8 + t0_1_127);
t0_2_63 += (1 - rt0_2_63) * t0_1_126;

int rt0_3_0 = (input[i] > (15+offset0));
int t0_3_0 = rt0_3_0 * (16 + t0_2_1);
t0_3_0 += (1 - rt0_3_0) * t0_2_0;

int rt0_3_1 = (input[i] > (47+offset0));
int t0_3_1 = rt0_3_1 * (16 + t0_2_3);
t0_3_1 += (1 - rt0_3_1) * t0_2_2;

int rt0_3_2 = (input[i] > (79+offset0));
int t0_3_2 = rt0_3_2 * (16 + t0_2_5);
t0_3_2 += (1 - rt0_3_2) * t0_2_4;

int rt0_3_3 = (input[i] > (111+offset0));
int t0_3_3 = rt0_3_3 * (16 + t0_2_7);
t0_3_3 += (1 - rt0_3_3) * t0_2_6;

int rt0_3_4 = (input[i] > (143+offset0));
int t0_3_4 = rt0_3_4 * (16 + t0_2_9);
t0_3_4 += (1 - rt0_3_4) * t0_2_8;

int rt0_3_5 = (input[i] > (175+offset0));
int t0_3_5 = rt0_3_5 * (16 + t0_2_11);
t0_3_5 += (1 - rt0_3_5) * t0_2_10;

int rt0_3_6 = (input[i] > (207+offset0));
int t0_3_6 = rt0_3_6 * (16 + t0_2_13);
t0_3_6 += (1 - rt0_3_6) * t0_2_12;

int rt0_3_7 = (input[i] > (239+offset0));
int t0_3_7 = rt0_3_7 * (16 + t0_2_15);
t0_3_7 += (1 - rt0_3_7) * t0_2_14;

int rt0_3_8 = (input[i] > (271+offset0));
int t0_3_8 = rt0_3_8 * (16 + t0_2_17);
t0_3_8 += (1 - rt0_3_8) * t0_2_16;

int rt0_3_9 = (input[i] > (303+offset0));
int t0_3_9 = rt0_3_9 * (16 + t0_2_19);
t0_3_9 += (1 - rt0_3_9) * t0_2_18;

int rt0_3_10 = (input[i] > (335+offset0));
int t0_3_10 = rt0_3_10 * (16 + t0_2_21);
t0_3_10 += (1 - rt0_3_10) * t0_2_20;

int rt0_3_11 = (input[i] > (367+offset0));
int t0_3_11 = rt0_3_11 * (16 + t0_2_23);
t0_3_11 += (1 - rt0_3_11) * t0_2_22;

int rt0_3_12 = (input[i] > (399+offset0));
int t0_3_12 = rt0_3_12 * (16 + t0_2_25);
t0_3_12 += (1 - rt0_3_12) * t0_2_24;

int rt0_3_13 = (input[i] > (431+offset0));
int t0_3_13 = rt0_3_13 * (16 + t0_2_27);
t0_3_13 += (1 - rt0_3_13) * t0_2_26;

int rt0_3_14 = (input[i] > (463+offset0));
int t0_3_14 = rt0_3_14 * (16 + t0_2_29);
t0_3_14 += (1 - rt0_3_14) * t0_2_28;

int rt0_3_15 = (input[i] > (495+offset0));
int t0_3_15 = rt0_3_15 * (16 + t0_2_31);
t0_3_15 += (1 - rt0_3_15) * t0_2_30;

int rt0_3_16 = (input[i] > (527+offset0));
int t0_3_16 = rt0_3_16 * (16 + t0_2_33);
t0_3_16 += (1 - rt0_3_16) * t0_2_32;

int rt0_3_17 = (input[i] > (559+offset0));
int t0_3_17 = rt0_3_17 * (16 + t0_2_35);
t0_3_17 += (1 - rt0_3_17) * t0_2_34;

int rt0_3_18 = (input[i] > (591+offset0));
int t0_3_18 = rt0_3_18 * (16 + t0_2_37);
t0_3_18 += (1 - rt0_3_18) * t0_2_36;

int rt0_3_19 = (input[i] > (623+offset0));
int t0_3_19 = rt0_3_19 * (16 + t0_2_39);
t0_3_19 += (1 - rt0_3_19) * t0_2_38;

int rt0_3_20 = (input[i] > (655+offset0));
int t0_3_20 = rt0_3_20 * (16 + t0_2_41);
t0_3_20 += (1 - rt0_3_20) * t0_2_40;

int rt0_3_21 = (input[i] > (687+offset0));
int t0_3_21 = rt0_3_21 * (16 + t0_2_43);
t0_3_21 += (1 - rt0_3_21) * t0_2_42;

int rt0_3_22 = (input[i] > (719+offset0));
int t0_3_22 = rt0_3_22 * (16 + t0_2_45);
t0_3_22 += (1 - rt0_3_22) * t0_2_44;

int rt0_3_23 = (input[i] > (751+offset0));
int t0_3_23 = rt0_3_23 * (16 + t0_2_47);
t0_3_23 += (1 - rt0_3_23) * t0_2_46;

int rt0_3_24 = (input[i] > (783+offset0));
int t0_3_24 = rt0_3_24 * (16 + t0_2_49);
t0_3_24 += (1 - rt0_3_24) * t0_2_48;

int rt0_3_25 = (input[i] > (815+offset0));
int t0_3_25 = rt0_3_25 * (16 + t0_2_51);
t0_3_25 += (1 - rt0_3_25) * t0_2_50;

int rt0_3_26 = (input[i] > (847+offset0));
int t0_3_26 = rt0_3_26 * (16 + t0_2_53);
t0_3_26 += (1 - rt0_3_26) * t0_2_52;

int rt0_3_27 = (input[i] > (879+offset0));
int t0_3_27 = rt0_3_27 * (16 + t0_2_55);
t0_3_27 += (1 - rt0_3_27) * t0_2_54;

int rt0_3_28 = (input[i] > (911+offset0));
int t0_3_28 = rt0_3_28 * (16 + t0_2_57);
t0_3_28 += (1 - rt0_3_28) * t0_2_56;

int rt0_3_29 = (input[i] > (943+offset0));
int t0_3_29 = rt0_3_29 * (16 + t0_2_59);
t0_3_29 += (1 - rt0_3_29) * t0_2_58;

int rt0_3_30 = (input[i] > (975+offset0));
int t0_3_30 = rt0_3_30 * (16 + t0_2_61);
t0_3_30 += (1 - rt0_3_30) * t0_2_60;

int rt0_3_31 = (input[i] > (1007+offset0));
int t0_3_31 = rt0_3_31 * (16 + t0_2_63);
t0_3_31 += (1 - rt0_3_31) * t0_2_62;

int rt0_4_0 = (input[i] > (31+offset0));
int t0_4_0 = rt0_4_0 * (32 + t0_3_1);
t0_4_0 += (1 - rt0_4_0) * t0_3_0;

int rt0_4_1 = (input[i] > (95+offset0));
int t0_4_1 = rt0_4_1 * (32 + t0_3_3);
t0_4_1 += (1 - rt0_4_1) * t0_3_2;

int rt0_4_2 = (input[i] > (159+offset0));
int t0_4_2 = rt0_4_2 * (32 + t0_3_5);
t0_4_2 += (1 - rt0_4_2) * t0_3_4;

int rt0_4_3 = (input[i] > (223+offset0));
int t0_4_3 = rt0_4_3 * (32 + t0_3_7);
t0_4_3 += (1 - rt0_4_3) * t0_3_6;

int rt0_4_4 = (input[i] > (287+offset0));
int t0_4_4 = rt0_4_4 * (32 + t0_3_9);
t0_4_4 += (1 - rt0_4_4) * t0_3_8;

int rt0_4_5 = (input[i] > (351+offset0));
int t0_4_5 = rt0_4_5 * (32 + t0_3_11);
t0_4_5 += (1 - rt0_4_5) * t0_3_10;

int rt0_4_6 = (input[i] > (415+offset0));
int t0_4_6 = rt0_4_6 * (32 + t0_3_13);
t0_4_6 += (1 - rt0_4_6) * t0_3_12;

int rt0_4_7 = (input[i] > (479+offset0));
int t0_4_7 = rt0_4_7 * (32 + t0_3_15);
t0_4_7 += (1 - rt0_4_7) * t0_3_14;

int rt0_4_8 = (input[i] > (543+offset0));
int t0_4_8 = rt0_4_8 * (32 + t0_3_17);
t0_4_8 += (1 - rt0_4_8) * t0_3_16;

int rt0_4_9 = (input[i] > (607+offset0));
int t0_4_9 = rt0_4_9 * (32 + t0_3_19);
t0_4_9 += (1 - rt0_4_9) * t0_3_18;

int rt0_4_10 = (input[i] > (671+offset0));
int t0_4_10 = rt0_4_10 * (32 + t0_3_21);
t0_4_10 += (1 - rt0_4_10) * t0_3_20;

int rt0_4_11 = (input[i] > (735+offset0));
int t0_4_11 = rt0_4_11 * (32 + t0_3_23);
t0_4_11 += (1 - rt0_4_11) * t0_3_22;

int rt0_4_12 = (input[i] > (799+offset0));
int t0_4_12 = rt0_4_12 * (32 + t0_3_25);
t0_4_12 += (1 - rt0_4_12) * t0_3_24;

int rt0_4_13 = (input[i] > (863+offset0));
int t0_4_13 = rt0_4_13 * (32 + t0_3_27);
t0_4_13 += (1 - rt0_4_13) * t0_3_26;

int rt0_4_14 = (input[i] > (927+offset0));
int t0_4_14 = rt0_4_14 * (32 + t0_3_29);
t0_4_14 += (1 - rt0_4_14) * t0_3_28;

int rt0_4_15 = (input[i] > (991+offset0));
int t0_4_15 = rt0_4_15 * (32 + t0_3_31);
t0_4_15 += (1 - rt0_4_15) * t0_3_30;

int rt0_5_0 = (input[i] > (63+offset0));
int t0_5_0 = rt0_5_0 * (64 + t0_4_1);
t0_5_0 += (1 - rt0_5_0) * t0_4_0;

int rt0_5_1 = (input[i] > (191+offset0));
int t0_5_1 = rt0_5_1 * (64 + t0_4_3);
t0_5_1 += (1 - rt0_5_1) * t0_4_2;

int rt0_5_2 = (input[i] > (319+offset0));
int t0_5_2 = rt0_5_2 * (64 + t0_4_5);
t0_5_2 += (1 - rt0_5_2) * t0_4_4;

int rt0_5_3 = (input[i] > (447+offset0));
int t0_5_3 = rt0_5_3 * (64 + t0_4_7);
t0_5_3 += (1 - rt0_5_3) * t0_4_6;

int rt0_5_4 = (input[i] > (575+offset0));
int t0_5_4 = rt0_5_4 * (64 + t0_4_9);
t0_5_4 += (1 - rt0_5_4) * t0_4_8;

int rt0_5_5 = (input[i] > (703+offset0));
int t0_5_5 = rt0_5_5 * (64 + t0_4_11);
t0_5_5 += (1 - rt0_5_5) * t0_4_10;

int rt0_5_6 = (input[i] > (831+offset0));
int t0_5_6 = rt0_5_6 * (64 + t0_4_13);
t0_5_6 += (1 - rt0_5_6) * t0_4_12;

int rt0_5_7 = (input[i] > (959+offset0));
int t0_5_7 = rt0_5_7 * (64 + t0_4_15);
t0_5_7 += (1 - rt0_5_7) * t0_4_14;

int rt0_6_0 = (input[i] > (127+offset0));
int t0_6_0 = rt0_6_0 * (128 + t0_5_1);
t0_6_0 += (1 - rt0_6_0) * t0_5_0;

int rt0_6_1 = (input[i] > (383+offset0));
int t0_6_1 = rt0_6_1 * (128 + t0_5_3);
t0_6_1 += (1 - rt0_6_1) * t0_5_2;

int rt0_6_2 = (input[i] > (639+offset0));
int t0_6_2 = rt0_6_2 * (128 + t0_5_5);
t0_6_2 += (1 - rt0_6_2) * t0_5_4;

int rt0_6_3 = (input[i] > (895+offset0));
int t0_6_3 = rt0_6_3 * (128 + t0_5_7);
t0_6_3 += (1 - rt0_6_3) * t0_5_6;

int rt0_7_0 = (input[i] > (255+offset0));
int t0_7_0 = rt0_7_0 * (256 + t0_6_1);
t0_7_0 += (1 - rt0_7_0) * t0_6_0;

int rt0_7_1 = (input[i] > (767+offset0));
int t0_7_1 = rt0_7_1 * (256 + t0_6_3);
t0_7_1 += (1 - rt0_7_1) * t0_6_2;

int rt0_8_0 = (input[i] > (511+offset0));
int t0_8_0 = rt0_8_0 * (512 + t0_7_1);
t0_8_0 += (1 - rt0_8_0) * t0_7_0;

int rt1_0_0 = (input[i] > (1+offset1));
int t1_0_0 = rt1_0_0 * (2 + (input[i] > (2+offset1)));
t1_0_0 += (1 - rt1_0_0) * (input[i] > (0+offset1));

int rt1_0_1 = (input[i] > (5+offset1));
int t1_0_1 = rt1_0_1 * (2 + (input[i] > (6+offset1)));
t1_0_1 += (1 - rt1_0_1) * (input[i] > (4+offset1));

int rt1_0_2 = (input[i] > (9+offset1));
int t1_0_2 = rt1_0_2 * (2 + (input[i] > (10+offset1)));
t1_0_2 += (1 - rt1_0_2) * (input[i] > (8+offset1));

int rt1_0_3 = (input[i] > (13+offset1));
int t1_0_3 = rt1_0_3 * (2 + (input[i] > (14+offset1)));
t1_0_3 += (1 - rt1_0_3) * (input[i] > (12+offset1));

int rt1_0_4 = (input[i] > (17+offset1));
int t1_0_4 = rt1_0_4 * (2 + (input[i] > (18+offset1)));
t1_0_4 += (1 - rt1_0_4) * (input[i] > (16+offset1));

int rt1_0_5 = (input[i] > (21+offset1));
int t1_0_5 = rt1_0_5 * (2 + (input[i] > (22+offset1)));
t1_0_5 += (1 - rt1_0_5) * (input[i] > (20+offset1));

int rt1_0_6 = (input[i] > (25+offset1));
int t1_0_6 = rt1_0_6 * (2 + (input[i] > (26+offset1)));
t1_0_6 += (1 - rt1_0_6) * (input[i] > (24+offset1));

int rt1_0_7 = (input[i] > (29+offset1));
int t1_0_7 = rt1_0_7 * (2 + (input[i] > (30+offset1)));
t1_0_7 += (1 - rt1_0_7) * (input[i] > (28+offset1));

int rt1_0_8 = (input[i] > (33+offset1));
int t1_0_8 = rt1_0_8 * (2 + (input[i] > (34+offset1)));
t1_0_8 += (1 - rt1_0_8) * (input[i] > (32+offset1));

int rt1_0_9 = (input[i] > (37+offset1));
int t1_0_9 = rt1_0_9 * (2 + (input[i] > (38+offset1)));
t1_0_9 += (1 - rt1_0_9) * (input[i] > (36+offset1));

int rt1_0_10 = (input[i] > (41+offset1));
int t1_0_10 = rt1_0_10 * (2 + (input[i] > (42+offset1)));
t1_0_10 += (1 - rt1_0_10) * (input[i] > (40+offset1));

int rt1_0_11 = (input[i] > (45+offset1));
int t1_0_11 = rt1_0_11 * (2 + (input[i] > (46+offset1)));
t1_0_11 += (1 - rt1_0_11) * (input[i] > (44+offset1));

int rt1_0_12 = (input[i] > (49+offset1));
int t1_0_12 = rt1_0_12 * (2 + (input[i] > (50+offset1)));
t1_0_12 += (1 - rt1_0_12) * (input[i] > (48+offset1));

int rt1_0_13 = (input[i] > (53+offset1));
int t1_0_13 = rt1_0_13 * (2 + (input[i] > (54+offset1)));
t1_0_13 += (1 - rt1_0_13) * (input[i] > (52+offset1));

int rt1_0_14 = (input[i] > (57+offset1));
int t1_0_14 = rt1_0_14 * (2 + (input[i] > (58+offset1)));
t1_0_14 += (1 - rt1_0_14) * (input[i] > (56+offset1));

int rt1_0_15 = (input[i] > (61+offset1));
int t1_0_15 = rt1_0_15 * (2 + (input[i] > (62+offset1)));
t1_0_15 += (1 - rt1_0_15) * (input[i] > (60+offset1));

int rt1_0_16 = (input[i] > (65+offset1));
int t1_0_16 = rt1_0_16 * (2 + (input[i] > (66+offset1)));
t1_0_16 += (1 - rt1_0_16) * (input[i] > (64+offset1));

int rt1_0_17 = (input[i] > (69+offset1));
int t1_0_17 = rt1_0_17 * (2 + (input[i] > (70+offset1)));
t1_0_17 += (1 - rt1_0_17) * (input[i] > (68+offset1));

int rt1_0_18 = (input[i] > (73+offset1));
int t1_0_18 = rt1_0_18 * (2 + (input[i] > (74+offset1)));
t1_0_18 += (1 - rt1_0_18) * (input[i] > (72+offset1));

int rt1_0_19 = (input[i] > (77+offset1));
int t1_0_19 = rt1_0_19 * (2 + (input[i] > (78+offset1)));
t1_0_19 += (1 - rt1_0_19) * (input[i] > (76+offset1));

int rt1_0_20 = (input[i] > (81+offset1));
int t1_0_20 = rt1_0_20 * (2 + (input[i] > (82+offset1)));
t1_0_20 += (1 - rt1_0_20) * (input[i] > (80+offset1));

int rt1_0_21 = (input[i] > (85+offset1));
int t1_0_21 = rt1_0_21 * (2 + (input[i] > (86+offset1)));
t1_0_21 += (1 - rt1_0_21) * (input[i] > (84+offset1));

int rt1_0_22 = (input[i] > (89+offset1));
int t1_0_22 = rt1_0_22 * (2 + (input[i] > (90+offset1)));
t1_0_22 += (1 - rt1_0_22) * (input[i] > (88+offset1));

int rt1_0_23 = (input[i] > (93+offset1));
int t1_0_23 = rt1_0_23 * (2 + (input[i] > (94+offset1)));
t1_0_23 += (1 - rt1_0_23) * (input[i] > (92+offset1));

int rt1_0_24 = (input[i] > (97+offset1));
int t1_0_24 = rt1_0_24 * (2 + (input[i] > (98+offset1)));
t1_0_24 += (1 - rt1_0_24) * (input[i] > (96+offset1));

int rt1_0_25 = (input[i] > (101+offset1));
int t1_0_25 = rt1_0_25 * (2 + (input[i] > (102+offset1)));
t1_0_25 += (1 - rt1_0_25) * (input[i] > (100+offset1));

int rt1_0_26 = (input[i] > (105+offset1));
int t1_0_26 = rt1_0_26 * (2 + (input[i] > (106+offset1)));
t1_0_26 += (1 - rt1_0_26) * (input[i] > (104+offset1));

int rt1_0_27 = (input[i] > (109+offset1));
int t1_0_27 = rt1_0_27 * (2 + (input[i] > (110+offset1)));
t1_0_27 += (1 - rt1_0_27) * (input[i] > (108+offset1));

int rt1_0_28 = (input[i] > (113+offset1));
int t1_0_28 = rt1_0_28 * (2 + (input[i] > (114+offset1)));
t1_0_28 += (1 - rt1_0_28) * (input[i] > (112+offset1));

int rt1_0_29 = (input[i] > (117+offset1));
int t1_0_29 = rt1_0_29 * (2 + (input[i] > (118+offset1)));
t1_0_29 += (1 - rt1_0_29) * (input[i] > (116+offset1));

int rt1_0_30 = (input[i] > (121+offset1));
int t1_0_30 = rt1_0_30 * (2 + (input[i] > (122+offset1)));
t1_0_30 += (1 - rt1_0_30) * (input[i] > (120+offset1));

int rt1_0_31 = (input[i] > (125+offset1));
int t1_0_31 = rt1_0_31 * (2 + (input[i] > (126+offset1)));
t1_0_31 += (1 - rt1_0_31) * (input[i] > (124+offset1));

int rt1_0_32 = (input[i] > (129+offset1));
int t1_0_32 = rt1_0_32 * (2 + (input[i] > (130+offset1)));
t1_0_32 += (1 - rt1_0_32) * (input[i] > (128+offset1));

int rt1_0_33 = (input[i] > (133+offset1));
int t1_0_33 = rt1_0_33 * (2 + (input[i] > (134+offset1)));
t1_0_33 += (1 - rt1_0_33) * (input[i] > (132+offset1));

int rt1_0_34 = (input[i] > (137+offset1));
int t1_0_34 = rt1_0_34 * (2 + (input[i] > (138+offset1)));
t1_0_34 += (1 - rt1_0_34) * (input[i] > (136+offset1));

int rt1_0_35 = (input[i] > (141+offset1));
int t1_0_35 = rt1_0_35 * (2 + (input[i] > (142+offset1)));
t1_0_35 += (1 - rt1_0_35) * (input[i] > (140+offset1));

int rt1_0_36 = (input[i] > (145+offset1));
int t1_0_36 = rt1_0_36 * (2 + (input[i] > (146+offset1)));
t1_0_36 += (1 - rt1_0_36) * (input[i] > (144+offset1));

int rt1_0_37 = (input[i] > (149+offset1));
int t1_0_37 = rt1_0_37 * (2 + (input[i] > (150+offset1)));
t1_0_37 += (1 - rt1_0_37) * (input[i] > (148+offset1));

int rt1_0_38 = (input[i] > (153+offset1));
int t1_0_38 = rt1_0_38 * (2 + (input[i] > (154+offset1)));
t1_0_38 += (1 - rt1_0_38) * (input[i] > (152+offset1));

int rt1_0_39 = (input[i] > (157+offset1));
int t1_0_39 = rt1_0_39 * (2 + (input[i] > (158+offset1)));
t1_0_39 += (1 - rt1_0_39) * (input[i] > (156+offset1));

int rt1_0_40 = (input[i] > (161+offset1));
int t1_0_40 = rt1_0_40 * (2 + (input[i] > (162+offset1)));
t1_0_40 += (1 - rt1_0_40) * (input[i] > (160+offset1));

int rt1_0_41 = (input[i] > (165+offset1));
int t1_0_41 = rt1_0_41 * (2 + (input[i] > (166+offset1)));
t1_0_41 += (1 - rt1_0_41) * (input[i] > (164+offset1));

int rt1_0_42 = (input[i] > (169+offset1));
int t1_0_42 = rt1_0_42 * (2 + (input[i] > (170+offset1)));
t1_0_42 += (1 - rt1_0_42) * (input[i] > (168+offset1));

int rt1_0_43 = (input[i] > (173+offset1));
int t1_0_43 = rt1_0_43 * (2 + (input[i] > (174+offset1)));
t1_0_43 += (1 - rt1_0_43) * (input[i] > (172+offset1));

int rt1_0_44 = (input[i] > (177+offset1));
int t1_0_44 = rt1_0_44 * (2 + (input[i] > (178+offset1)));
t1_0_44 += (1 - rt1_0_44) * (input[i] > (176+offset1));

int rt1_0_45 = (input[i] > (181+offset1));
int t1_0_45 = rt1_0_45 * (2 + (input[i] > (182+offset1)));
t1_0_45 += (1 - rt1_0_45) * (input[i] > (180+offset1));

int rt1_0_46 = (input[i] > (185+offset1));
int t1_0_46 = rt1_0_46 * (2 + (input[i] > (186+offset1)));
t1_0_46 += (1 - rt1_0_46) * (input[i] > (184+offset1));

int rt1_0_47 = (input[i] > (189+offset1));
int t1_0_47 = rt1_0_47 * (2 + (input[i] > (190+offset1)));
t1_0_47 += (1 - rt1_0_47) * (input[i] > (188+offset1));

int rt1_0_48 = (input[i] > (193+offset1));
int t1_0_48 = rt1_0_48 * (2 + (input[i] > (194+offset1)));
t1_0_48 += (1 - rt1_0_48) * (input[i] > (192+offset1));

int rt1_0_49 = (input[i] > (197+offset1));
int t1_0_49 = rt1_0_49 * (2 + (input[i] > (198+offset1)));
t1_0_49 += (1 - rt1_0_49) * (input[i] > (196+offset1));

int rt1_0_50 = (input[i] > (201+offset1));
int t1_0_50 = rt1_0_50 * (2 + (input[i] > (202+offset1)));
t1_0_50 += (1 - rt1_0_50) * (input[i] > (200+offset1));

int rt1_0_51 = (input[i] > (205+offset1));
int t1_0_51 = rt1_0_51 * (2 + (input[i] > (206+offset1)));
t1_0_51 += (1 - rt1_0_51) * (input[i] > (204+offset1));

int rt1_0_52 = (input[i] > (209+offset1));
int t1_0_52 = rt1_0_52 * (2 + (input[i] > (210+offset1)));
t1_0_52 += (1 - rt1_0_52) * (input[i] > (208+offset1));

int rt1_0_53 = (input[i] > (213+offset1));
int t1_0_53 = rt1_0_53 * (2 + (input[i] > (214+offset1)));
t1_0_53 += (1 - rt1_0_53) * (input[i] > (212+offset1));

int rt1_0_54 = (input[i] > (217+offset1));
int t1_0_54 = rt1_0_54 * (2 + (input[i] > (218+offset1)));
t1_0_54 += (1 - rt1_0_54) * (input[i] > (216+offset1));

int rt1_0_55 = (input[i] > (221+offset1));
int t1_0_55 = rt1_0_55 * (2 + (input[i] > (222+offset1)));
t1_0_55 += (1 - rt1_0_55) * (input[i] > (220+offset1));

int rt1_0_56 = (input[i] > (225+offset1));
int t1_0_56 = rt1_0_56 * (2 + (input[i] > (226+offset1)));
t1_0_56 += (1 - rt1_0_56) * (input[i] > (224+offset1));

int rt1_0_57 = (input[i] > (229+offset1));
int t1_0_57 = rt1_0_57 * (2 + (input[i] > (230+offset1)));
t1_0_57 += (1 - rt1_0_57) * (input[i] > (228+offset1));

int rt1_0_58 = (input[i] > (233+offset1));
int t1_0_58 = rt1_0_58 * (2 + (input[i] > (234+offset1)));
t1_0_58 += (1 - rt1_0_58) * (input[i] > (232+offset1));

int rt1_0_59 = (input[i] > (237+offset1));
int t1_0_59 = rt1_0_59 * (2 + (input[i] > (238+offset1)));
t1_0_59 += (1 - rt1_0_59) * (input[i] > (236+offset1));

int rt1_0_60 = (input[i] > (241+offset1));
int t1_0_60 = rt1_0_60 * (2 + (input[i] > (242+offset1)));
t1_0_60 += (1 - rt1_0_60) * (input[i] > (240+offset1));

int rt1_0_61 = (input[i] > (245+offset1));
int t1_0_61 = rt1_0_61 * (2 + (input[i] > (246+offset1)));
t1_0_61 += (1 - rt1_0_61) * (input[i] > (244+offset1));

int rt1_0_62 = (input[i] > (249+offset1));
int t1_0_62 = rt1_0_62 * (2 + (input[i] > (250+offset1)));
t1_0_62 += (1 - rt1_0_62) * (input[i] > (248+offset1));

int rt1_0_63 = (input[i] > (253+offset1));
int t1_0_63 = rt1_0_63 * (2 + (input[i] > (254+offset1)));
t1_0_63 += (1 - rt1_0_63) * (input[i] > (252+offset1));

int rt1_0_64 = (input[i] > (257+offset1));
int t1_0_64 = rt1_0_64 * (2 + (input[i] > (258+offset1)));
t1_0_64 += (1 - rt1_0_64) * (input[i] > (256+offset1));

int rt1_0_65 = (input[i] > (261+offset1));
int t1_0_65 = rt1_0_65 * (2 + (input[i] > (262+offset1)));
t1_0_65 += (1 - rt1_0_65) * (input[i] > (260+offset1));

int rt1_0_66 = (input[i] > (265+offset1));
int t1_0_66 = rt1_0_66 * (2 + (input[i] > (266+offset1)));
t1_0_66 += (1 - rt1_0_66) * (input[i] > (264+offset1));

int rt1_0_67 = (input[i] > (269+offset1));
int t1_0_67 = rt1_0_67 * (2 + (input[i] > (270+offset1)));
t1_0_67 += (1 - rt1_0_67) * (input[i] > (268+offset1));

int rt1_0_68 = (input[i] > (273+offset1));
int t1_0_68 = rt1_0_68 * (2 + (input[i] > (274+offset1)));
t1_0_68 += (1 - rt1_0_68) * (input[i] > (272+offset1));

int rt1_0_69 = (input[i] > (277+offset1));
int t1_0_69 = rt1_0_69 * (2 + (input[i] > (278+offset1)));
t1_0_69 += (1 - rt1_0_69) * (input[i] > (276+offset1));

int rt1_0_70 = (input[i] > (281+offset1));
int t1_0_70 = rt1_0_70 * (2 + (input[i] > (282+offset1)));
t1_0_70 += (1 - rt1_0_70) * (input[i] > (280+offset1));

int rt1_0_71 = (input[i] > (285+offset1));
int t1_0_71 = rt1_0_71 * (2 + (input[i] > (286+offset1)));
t1_0_71 += (1 - rt1_0_71) * (input[i] > (284+offset1));

int rt1_0_72 = (input[i] > (289+offset1));
int t1_0_72 = rt1_0_72 * (2 + (input[i] > (290+offset1)));
t1_0_72 += (1 - rt1_0_72) * (input[i] > (288+offset1));

int rt1_0_73 = (input[i] > (293+offset1));
int t1_0_73 = rt1_0_73 * (2 + (input[i] > (294+offset1)));
t1_0_73 += (1 - rt1_0_73) * (input[i] > (292+offset1));

int rt1_0_74 = (input[i] > (297+offset1));
int t1_0_74 = rt1_0_74 * (2 + (input[i] > (298+offset1)));
t1_0_74 += (1 - rt1_0_74) * (input[i] > (296+offset1));

int rt1_0_75 = (input[i] > (301+offset1));
int t1_0_75 = rt1_0_75 * (2 + (input[i] > (302+offset1)));
t1_0_75 += (1 - rt1_0_75) * (input[i] > (300+offset1));

int rt1_0_76 = (input[i] > (305+offset1));
int t1_0_76 = rt1_0_76 * (2 + (input[i] > (306+offset1)));
t1_0_76 += (1 - rt1_0_76) * (input[i] > (304+offset1));

int rt1_0_77 = (input[i] > (309+offset1));
int t1_0_77 = rt1_0_77 * (2 + (input[i] > (310+offset1)));
t1_0_77 += (1 - rt1_0_77) * (input[i] > (308+offset1));

int rt1_0_78 = (input[i] > (313+offset1));
int t1_0_78 = rt1_0_78 * (2 + (input[i] > (314+offset1)));
t1_0_78 += (1 - rt1_0_78) * (input[i] > (312+offset1));

int rt1_0_79 = (input[i] > (317+offset1));
int t1_0_79 = rt1_0_79 * (2 + (input[i] > (318+offset1)));
t1_0_79 += (1 - rt1_0_79) * (input[i] > (316+offset1));

int rt1_0_80 = (input[i] > (321+offset1));
int t1_0_80 = rt1_0_80 * (2 + (input[i] > (322+offset1)));
t1_0_80 += (1 - rt1_0_80) * (input[i] > (320+offset1));

int rt1_0_81 = (input[i] > (325+offset1));
int t1_0_81 = rt1_0_81 * (2 + (input[i] > (326+offset1)));
t1_0_81 += (1 - rt1_0_81) * (input[i] > (324+offset1));

int rt1_0_82 = (input[i] > (329+offset1));
int t1_0_82 = rt1_0_82 * (2 + (input[i] > (330+offset1)));
t1_0_82 += (1 - rt1_0_82) * (input[i] > (328+offset1));

int rt1_0_83 = (input[i] > (333+offset1));
int t1_0_83 = rt1_0_83 * (2 + (input[i] > (334+offset1)));
t1_0_83 += (1 - rt1_0_83) * (input[i] > (332+offset1));

int rt1_0_84 = (input[i] > (337+offset1));
int t1_0_84 = rt1_0_84 * (2 + (input[i] > (338+offset1)));
t1_0_84 += (1 - rt1_0_84) * (input[i] > (336+offset1));

int rt1_0_85 = (input[i] > (341+offset1));
int t1_0_85 = rt1_0_85 * (2 + (input[i] > (342+offset1)));
t1_0_85 += (1 - rt1_0_85) * (input[i] > (340+offset1));

int rt1_0_86 = (input[i] > (345+offset1));
int t1_0_86 = rt1_0_86 * (2 + (input[i] > (346+offset1)));
t1_0_86 += (1 - rt1_0_86) * (input[i] > (344+offset1));

int rt1_0_87 = (input[i] > (349+offset1));
int t1_0_87 = rt1_0_87 * (2 + (input[i] > (350+offset1)));
t1_0_87 += (1 - rt1_0_87) * (input[i] > (348+offset1));

int rt1_0_88 = (input[i] > (353+offset1));
int t1_0_88 = rt1_0_88 * (2 + (input[i] > (354+offset1)));
t1_0_88 += (1 - rt1_0_88) * (input[i] > (352+offset1));

int rt1_0_89 = (input[i] > (357+offset1));
int t1_0_89 = rt1_0_89 * (2 + (input[i] > (358+offset1)));
t1_0_89 += (1 - rt1_0_89) * (input[i] > (356+offset1));

int rt1_0_90 = (input[i] > (361+offset1));
int t1_0_90 = rt1_0_90 * (2 + (input[i] > (362+offset1)));
t1_0_90 += (1 - rt1_0_90) * (input[i] > (360+offset1));

int rt1_0_91 = (input[i] > (365+offset1));
int t1_0_91 = rt1_0_91 * (2 + (input[i] > (366+offset1)));
t1_0_91 += (1 - rt1_0_91) * (input[i] > (364+offset1));

int rt1_0_92 = (input[i] > (369+offset1));
int t1_0_92 = rt1_0_92 * (2 + (input[i] > (370+offset1)));
t1_0_92 += (1 - rt1_0_92) * (input[i] > (368+offset1));

int rt1_0_93 = (input[i] > (373+offset1));
int t1_0_93 = rt1_0_93 * (2 + (input[i] > (374+offset1)));
t1_0_93 += (1 - rt1_0_93) * (input[i] > (372+offset1));

int rt1_0_94 = (input[i] > (377+offset1));
int t1_0_94 = rt1_0_94 * (2 + (input[i] > (378+offset1)));
t1_0_94 += (1 - rt1_0_94) * (input[i] > (376+offset1));

int rt1_0_95 = (input[i] > (381+offset1));
int t1_0_95 = rt1_0_95 * (2 + (input[i] > (382+offset1)));
t1_0_95 += (1 - rt1_0_95) * (input[i] > (380+offset1));

int rt1_0_96 = (input[i] > (385+offset1));
int t1_0_96 = rt1_0_96 * (2 + (input[i] > (386+offset1)));
t1_0_96 += (1 - rt1_0_96) * (input[i] > (384+offset1));

int rt1_0_97 = (input[i] > (389+offset1));
int t1_0_97 = rt1_0_97 * (2 + (input[i] > (390+offset1)));
t1_0_97 += (1 - rt1_0_97) * (input[i] > (388+offset1));

int rt1_0_98 = (input[i] > (393+offset1));
int t1_0_98 = rt1_0_98 * (2 + (input[i] > (394+offset1)));
t1_0_98 += (1 - rt1_0_98) * (input[i] > (392+offset1));

int rt1_0_99 = (input[i] > (397+offset1));
int t1_0_99 = rt1_0_99 * (2 + (input[i] > (398+offset1)));
t1_0_99 += (1 - rt1_0_99) * (input[i] > (396+offset1));

int rt1_0_100 = (input[i] > (401+offset1));
int t1_0_100 = rt1_0_100 * (2 + (input[i] > (402+offset1)));
t1_0_100 += (1 - rt1_0_100) * (input[i] > (400+offset1));

int rt1_0_101 = (input[i] > (405+offset1));
int t1_0_101 = rt1_0_101 * (2 + (input[i] > (406+offset1)));
t1_0_101 += (1 - rt1_0_101) * (input[i] > (404+offset1));

int rt1_0_102 = (input[i] > (409+offset1));
int t1_0_102 = rt1_0_102 * (2 + (input[i] > (410+offset1)));
t1_0_102 += (1 - rt1_0_102) * (input[i] > (408+offset1));

int rt1_0_103 = (input[i] > (413+offset1));
int t1_0_103 = rt1_0_103 * (2 + (input[i] > (414+offset1)));
t1_0_103 += (1 - rt1_0_103) * (input[i] > (412+offset1));

int rt1_0_104 = (input[i] > (417+offset1));
int t1_0_104 = rt1_0_104 * (2 + (input[i] > (418+offset1)));
t1_0_104 += (1 - rt1_0_104) * (input[i] > (416+offset1));

int rt1_0_105 = (input[i] > (421+offset1));
int t1_0_105 = rt1_0_105 * (2 + (input[i] > (422+offset1)));
t1_0_105 += (1 - rt1_0_105) * (input[i] > (420+offset1));

int rt1_0_106 = (input[i] > (425+offset1));
int t1_0_106 = rt1_0_106 * (2 + (input[i] > (426+offset1)));
t1_0_106 += (1 - rt1_0_106) * (input[i] > (424+offset1));

int rt1_0_107 = (input[i] > (429+offset1));
int t1_0_107 = rt1_0_107 * (2 + (input[i] > (430+offset1)));
t1_0_107 += (1 - rt1_0_107) * (input[i] > (428+offset1));

int rt1_0_108 = (input[i] > (433+offset1));
int t1_0_108 = rt1_0_108 * (2 + (input[i] > (434+offset1)));
t1_0_108 += (1 - rt1_0_108) * (input[i] > (432+offset1));

int rt1_0_109 = (input[i] > (437+offset1));
int t1_0_109 = rt1_0_109 * (2 + (input[i] > (438+offset1)));
t1_0_109 += (1 - rt1_0_109) * (input[i] > (436+offset1));

int rt1_0_110 = (input[i] > (441+offset1));
int t1_0_110 = rt1_0_110 * (2 + (input[i] > (442+offset1)));
t1_0_110 += (1 - rt1_0_110) * (input[i] > (440+offset1));

int rt1_0_111 = (input[i] > (445+offset1));
int t1_0_111 = rt1_0_111 * (2 + (input[i] > (446+offset1)));
t1_0_111 += (1 - rt1_0_111) * (input[i] > (444+offset1));

int rt1_0_112 = (input[i] > (449+offset1));
int t1_0_112 = rt1_0_112 * (2 + (input[i] > (450+offset1)));
t1_0_112 += (1 - rt1_0_112) * (input[i] > (448+offset1));

int rt1_0_113 = (input[i] > (453+offset1));
int t1_0_113 = rt1_0_113 * (2 + (input[i] > (454+offset1)));
t1_0_113 += (1 - rt1_0_113) * (input[i] > (452+offset1));

int rt1_0_114 = (input[i] > (457+offset1));
int t1_0_114 = rt1_0_114 * (2 + (input[i] > (458+offset1)));
t1_0_114 += (1 - rt1_0_114) * (input[i] > (456+offset1));

int rt1_0_115 = (input[i] > (461+offset1));
int t1_0_115 = rt1_0_115 * (2 + (input[i] > (462+offset1)));
t1_0_115 += (1 - rt1_0_115) * (input[i] > (460+offset1));

int rt1_0_116 = (input[i] > (465+offset1));
int t1_0_116 = rt1_0_116 * (2 + (input[i] > (466+offset1)));
t1_0_116 += (1 - rt1_0_116) * (input[i] > (464+offset1));

int rt1_0_117 = (input[i] > (469+offset1));
int t1_0_117 = rt1_0_117 * (2 + (input[i] > (470+offset1)));
t1_0_117 += (1 - rt1_0_117) * (input[i] > (468+offset1));

int rt1_0_118 = (input[i] > (473+offset1));
int t1_0_118 = rt1_0_118 * (2 + (input[i] > (474+offset1)));
t1_0_118 += (1 - rt1_0_118) * (input[i] > (472+offset1));

int rt1_0_119 = (input[i] > (477+offset1));
int t1_0_119 = rt1_0_119 * (2 + (input[i] > (478+offset1)));
t1_0_119 += (1 - rt1_0_119) * (input[i] > (476+offset1));

int rt1_0_120 = (input[i] > (481+offset1));
int t1_0_120 = rt1_0_120 * (2 + (input[i] > (482+offset1)));
t1_0_120 += (1 - rt1_0_120) * (input[i] > (480+offset1));

int rt1_0_121 = (input[i] > (485+offset1));
int t1_0_121 = rt1_0_121 * (2 + (input[i] > (486+offset1)));
t1_0_121 += (1 - rt1_0_121) * (input[i] > (484+offset1));

int rt1_0_122 = (input[i] > (489+offset1));
int t1_0_122 = rt1_0_122 * (2 + (input[i] > (490+offset1)));
t1_0_122 += (1 - rt1_0_122) * (input[i] > (488+offset1));

int rt1_0_123 = (input[i] > (493+offset1));
int t1_0_123 = rt1_0_123 * (2 + (input[i] > (494+offset1)));
t1_0_123 += (1 - rt1_0_123) * (input[i] > (492+offset1));

int rt1_0_124 = (input[i] > (497+offset1));
int t1_0_124 = rt1_0_124 * (2 + (input[i] > (498+offset1)));
t1_0_124 += (1 - rt1_0_124) * (input[i] > (496+offset1));

int rt1_0_125 = (input[i] > (501+offset1));
int t1_0_125 = rt1_0_125 * (2 + (input[i] > (502+offset1)));
t1_0_125 += (1 - rt1_0_125) * (input[i] > (500+offset1));

int rt1_0_126 = (input[i] > (505+offset1));
int t1_0_126 = rt1_0_126 * (2 + (input[i] > (506+offset1)));
t1_0_126 += (1 - rt1_0_126) * (input[i] > (504+offset1));

int rt1_0_127 = (input[i] > (509+offset1));
int t1_0_127 = rt1_0_127 * (2 + (input[i] > (510+offset1)));
t1_0_127 += (1 - rt1_0_127) * (input[i] > (508+offset1));

int rt1_0_128 = (input[i] > (513+offset1));
int t1_0_128 = rt1_0_128 * (2 + (input[i] > (514+offset1)));
t1_0_128 += (1 - rt1_0_128) * (input[i] > (512+offset1));

int rt1_0_129 = (input[i] > (517+offset1));
int t1_0_129 = rt1_0_129 * (2 + (input[i] > (518+offset1)));
t1_0_129 += (1 - rt1_0_129) * (input[i] > (516+offset1));

int rt1_0_130 = (input[i] > (521+offset1));
int t1_0_130 = rt1_0_130 * (2 + (input[i] > (522+offset1)));
t1_0_130 += (1 - rt1_0_130) * (input[i] > (520+offset1));

int rt1_0_131 = (input[i] > (525+offset1));
int t1_0_131 = rt1_0_131 * (2 + (input[i] > (526+offset1)));
t1_0_131 += (1 - rt1_0_131) * (input[i] > (524+offset1));

int rt1_0_132 = (input[i] > (529+offset1));
int t1_0_132 = rt1_0_132 * (2 + (input[i] > (530+offset1)));
t1_0_132 += (1 - rt1_0_132) * (input[i] > (528+offset1));

int rt1_0_133 = (input[i] > (533+offset1));
int t1_0_133 = rt1_0_133 * (2 + (input[i] > (534+offset1)));
t1_0_133 += (1 - rt1_0_133) * (input[i] > (532+offset1));

int rt1_0_134 = (input[i] > (537+offset1));
int t1_0_134 = rt1_0_134 * (2 + (input[i] > (538+offset1)));
t1_0_134 += (1 - rt1_0_134) * (input[i] > (536+offset1));

int rt1_0_135 = (input[i] > (541+offset1));
int t1_0_135 = rt1_0_135 * (2 + (input[i] > (542+offset1)));
t1_0_135 += (1 - rt1_0_135) * (input[i] > (540+offset1));

int rt1_0_136 = (input[i] > (545+offset1));
int t1_0_136 = rt1_0_136 * (2 + (input[i] > (546+offset1)));
t1_0_136 += (1 - rt1_0_136) * (input[i] > (544+offset1));

int rt1_0_137 = (input[i] > (549+offset1));
int t1_0_137 = rt1_0_137 * (2 + (input[i] > (550+offset1)));
t1_0_137 += (1 - rt1_0_137) * (input[i] > (548+offset1));

int rt1_0_138 = (input[i] > (553+offset1));
int t1_0_138 = rt1_0_138 * (2 + (input[i] > (554+offset1)));
t1_0_138 += (1 - rt1_0_138) * (input[i] > (552+offset1));

int rt1_0_139 = (input[i] > (557+offset1));
int t1_0_139 = rt1_0_139 * (2 + (input[i] > (558+offset1)));
t1_0_139 += (1 - rt1_0_139) * (input[i] > (556+offset1));

int rt1_0_140 = (input[i] > (561+offset1));
int t1_0_140 = rt1_0_140 * (2 + (input[i] > (562+offset1)));
t1_0_140 += (1 - rt1_0_140) * (input[i] > (560+offset1));

int rt1_0_141 = (input[i] > (565+offset1));
int t1_0_141 = rt1_0_141 * (2 + (input[i] > (566+offset1)));
t1_0_141 += (1 - rt1_0_141) * (input[i] > (564+offset1));

int rt1_0_142 = (input[i] > (569+offset1));
int t1_0_142 = rt1_0_142 * (2 + (input[i] > (570+offset1)));
t1_0_142 += (1 - rt1_0_142) * (input[i] > (568+offset1));

int rt1_0_143 = (input[i] > (573+offset1));
int t1_0_143 = rt1_0_143 * (2 + (input[i] > (574+offset1)));
t1_0_143 += (1 - rt1_0_143) * (input[i] > (572+offset1));

int rt1_0_144 = (input[i] > (577+offset1));
int t1_0_144 = rt1_0_144 * (2 + (input[i] > (578+offset1)));
t1_0_144 += (1 - rt1_0_144) * (input[i] > (576+offset1));

int rt1_0_145 = (input[i] > (581+offset1));
int t1_0_145 = rt1_0_145 * (2 + (input[i] > (582+offset1)));
t1_0_145 += (1 - rt1_0_145) * (input[i] > (580+offset1));

int rt1_0_146 = (input[i] > (585+offset1));
int t1_0_146 = rt1_0_146 * (2 + (input[i] > (586+offset1)));
t1_0_146 += (1 - rt1_0_146) * (input[i] > (584+offset1));

int rt1_0_147 = (input[i] > (589+offset1));
int t1_0_147 = rt1_0_147 * (2 + (input[i] > (590+offset1)));
t1_0_147 += (1 - rt1_0_147) * (input[i] > (588+offset1));

int rt1_0_148 = (input[i] > (593+offset1));
int t1_0_148 = rt1_0_148 * (2 + (input[i] > (594+offset1)));
t1_0_148 += (1 - rt1_0_148) * (input[i] > (592+offset1));

int rt1_0_149 = (input[i] > (597+offset1));
int t1_0_149 = rt1_0_149 * (2 + (input[i] > (598+offset1)));
t1_0_149 += (1 - rt1_0_149) * (input[i] > (596+offset1));

int rt1_0_150 = (input[i] > (601+offset1));
int t1_0_150 = rt1_0_150 * (2 + (input[i] > (602+offset1)));
t1_0_150 += (1 - rt1_0_150) * (input[i] > (600+offset1));

int rt1_0_151 = (input[i] > (605+offset1));
int t1_0_151 = rt1_0_151 * (2 + (input[i] > (606+offset1)));
t1_0_151 += (1 - rt1_0_151) * (input[i] > (604+offset1));

int rt1_0_152 = (input[i] > (609+offset1));
int t1_0_152 = rt1_0_152 * (2 + (input[i] > (610+offset1)));
t1_0_152 += (1 - rt1_0_152) * (input[i] > (608+offset1));

int rt1_0_153 = (input[i] > (613+offset1));
int t1_0_153 = rt1_0_153 * (2 + (input[i] > (614+offset1)));
t1_0_153 += (1 - rt1_0_153) * (input[i] > (612+offset1));

int rt1_0_154 = (input[i] > (617+offset1));
int t1_0_154 = rt1_0_154 * (2 + (input[i] > (618+offset1)));
t1_0_154 += (1 - rt1_0_154) * (input[i] > (616+offset1));

int rt1_0_155 = (input[i] > (621+offset1));
int t1_0_155 = rt1_0_155 * (2 + (input[i] > (622+offset1)));
t1_0_155 += (1 - rt1_0_155) * (input[i] > (620+offset1));

int rt1_0_156 = (input[i] > (625+offset1));
int t1_0_156 = rt1_0_156 * (2 + (input[i] > (626+offset1)));
t1_0_156 += (1 - rt1_0_156) * (input[i] > (624+offset1));

int rt1_0_157 = (input[i] > (629+offset1));
int t1_0_157 = rt1_0_157 * (2 + (input[i] > (630+offset1)));
t1_0_157 += (1 - rt1_0_157) * (input[i] > (628+offset1));

int rt1_0_158 = (input[i] > (633+offset1));
int t1_0_158 = rt1_0_158 * (2 + (input[i] > (634+offset1)));
t1_0_158 += (1 - rt1_0_158) * (input[i] > (632+offset1));

int rt1_0_159 = (input[i] > (637+offset1));
int t1_0_159 = rt1_0_159 * (2 + (input[i] > (638+offset1)));
t1_0_159 += (1 - rt1_0_159) * (input[i] > (636+offset1));

int rt1_0_160 = (input[i] > (641+offset1));
int t1_0_160 = rt1_0_160 * (2 + (input[i] > (642+offset1)));
t1_0_160 += (1 - rt1_0_160) * (input[i] > (640+offset1));

int rt1_0_161 = (input[i] > (645+offset1));
int t1_0_161 = rt1_0_161 * (2 + (input[i] > (646+offset1)));
t1_0_161 += (1 - rt1_0_161) * (input[i] > (644+offset1));

int rt1_0_162 = (input[i] > (649+offset1));
int t1_0_162 = rt1_0_162 * (2 + (input[i] > (650+offset1)));
t1_0_162 += (1 - rt1_0_162) * (input[i] > (648+offset1));

int rt1_0_163 = (input[i] > (653+offset1));
int t1_0_163 = rt1_0_163 * (2 + (input[i] > (654+offset1)));
t1_0_163 += (1 - rt1_0_163) * (input[i] > (652+offset1));

int rt1_0_164 = (input[i] > (657+offset1));
int t1_0_164 = rt1_0_164 * (2 + (input[i] > (658+offset1)));
t1_0_164 += (1 - rt1_0_164) * (input[i] > (656+offset1));

int rt1_0_165 = (input[i] > (661+offset1));
int t1_0_165 = rt1_0_165 * (2 + (input[i] > (662+offset1)));
t1_0_165 += (1 - rt1_0_165) * (input[i] > (660+offset1));

int rt1_0_166 = (input[i] > (665+offset1));
int t1_0_166 = rt1_0_166 * (2 + (input[i] > (666+offset1)));
t1_0_166 += (1 - rt1_0_166) * (input[i] > (664+offset1));

int rt1_0_167 = (input[i] > (669+offset1));
int t1_0_167 = rt1_0_167 * (2 + (input[i] > (670+offset1)));
t1_0_167 += (1 - rt1_0_167) * (input[i] > (668+offset1));

int rt1_0_168 = (input[i] > (673+offset1));
int t1_0_168 = rt1_0_168 * (2 + (input[i] > (674+offset1)));
t1_0_168 += (1 - rt1_0_168) * (input[i] > (672+offset1));

int rt1_0_169 = (input[i] > (677+offset1));
int t1_0_169 = rt1_0_169 * (2 + (input[i] > (678+offset1)));
t1_0_169 += (1 - rt1_0_169) * (input[i] > (676+offset1));

int rt1_0_170 = (input[i] > (681+offset1));
int t1_0_170 = rt1_0_170 * (2 + (input[i] > (682+offset1)));
t1_0_170 += (1 - rt1_0_170) * (input[i] > (680+offset1));

int rt1_0_171 = (input[i] > (685+offset1));
int t1_0_171 = rt1_0_171 * (2 + (input[i] > (686+offset1)));
t1_0_171 += (1 - rt1_0_171) * (input[i] > (684+offset1));

int rt1_0_172 = (input[i] > (689+offset1));
int t1_0_172 = rt1_0_172 * (2 + (input[i] > (690+offset1)));
t1_0_172 += (1 - rt1_0_172) * (input[i] > (688+offset1));

int rt1_0_173 = (input[i] > (693+offset1));
int t1_0_173 = rt1_0_173 * (2 + (input[i] > (694+offset1)));
t1_0_173 += (1 - rt1_0_173) * (input[i] > (692+offset1));

int rt1_0_174 = (input[i] > (697+offset1));
int t1_0_174 = rt1_0_174 * (2 + (input[i] > (698+offset1)));
t1_0_174 += (1 - rt1_0_174) * (input[i] > (696+offset1));

int rt1_0_175 = (input[i] > (701+offset1));
int t1_0_175 = rt1_0_175 * (2 + (input[i] > (702+offset1)));
t1_0_175 += (1 - rt1_0_175) * (input[i] > (700+offset1));

int rt1_0_176 = (input[i] > (705+offset1));
int t1_0_176 = rt1_0_176 * (2 + (input[i] > (706+offset1)));
t1_0_176 += (1 - rt1_0_176) * (input[i] > (704+offset1));

int rt1_0_177 = (input[i] > (709+offset1));
int t1_0_177 = rt1_0_177 * (2 + (input[i] > (710+offset1)));
t1_0_177 += (1 - rt1_0_177) * (input[i] > (708+offset1));

int rt1_0_178 = (input[i] > (713+offset1));
int t1_0_178 = rt1_0_178 * (2 + (input[i] > (714+offset1)));
t1_0_178 += (1 - rt1_0_178) * (input[i] > (712+offset1));

int rt1_0_179 = (input[i] > (717+offset1));
int t1_0_179 = rt1_0_179 * (2 + (input[i] > (718+offset1)));
t1_0_179 += (1 - rt1_0_179) * (input[i] > (716+offset1));

int rt1_0_180 = (input[i] > (721+offset1));
int t1_0_180 = rt1_0_180 * (2 + (input[i] > (722+offset1)));
t1_0_180 += (1 - rt1_0_180) * (input[i] > (720+offset1));

int rt1_0_181 = (input[i] > (725+offset1));
int t1_0_181 = rt1_0_181 * (2 + (input[i] > (726+offset1)));
t1_0_181 += (1 - rt1_0_181) * (input[i] > (724+offset1));

int rt1_0_182 = (input[i] > (729+offset1));
int t1_0_182 = rt1_0_182 * (2 + (input[i] > (730+offset1)));
t1_0_182 += (1 - rt1_0_182) * (input[i] > (728+offset1));

int rt1_0_183 = (input[i] > (733+offset1));
int t1_0_183 = rt1_0_183 * (2 + (input[i] > (734+offset1)));
t1_0_183 += (1 - rt1_0_183) * (input[i] > (732+offset1));

int rt1_0_184 = (input[i] > (737+offset1));
int t1_0_184 = rt1_0_184 * (2 + (input[i] > (738+offset1)));
t1_0_184 += (1 - rt1_0_184) * (input[i] > (736+offset1));

int rt1_0_185 = (input[i] > (741+offset1));
int t1_0_185 = rt1_0_185 * (2 + (input[i] > (742+offset1)));
t1_0_185 += (1 - rt1_0_185) * (input[i] > (740+offset1));

int rt1_0_186 = (input[i] > (745+offset1));
int t1_0_186 = rt1_0_186 * (2 + (input[i] > (746+offset1)));
t1_0_186 += (1 - rt1_0_186) * (input[i] > (744+offset1));

int rt1_0_187 = (input[i] > (749+offset1));
int t1_0_187 = rt1_0_187 * (2 + (input[i] > (750+offset1)));
t1_0_187 += (1 - rt1_0_187) * (input[i] > (748+offset1));

int rt1_0_188 = (input[i] > (753+offset1));
int t1_0_188 = rt1_0_188 * (2 + (input[i] > (754+offset1)));
t1_0_188 += (1 - rt1_0_188) * (input[i] > (752+offset1));

int rt1_0_189 = (input[i] > (757+offset1));
int t1_0_189 = rt1_0_189 * (2 + (input[i] > (758+offset1)));
t1_0_189 += (1 - rt1_0_189) * (input[i] > (756+offset1));

int rt1_0_190 = (input[i] > (761+offset1));
int t1_0_190 = rt1_0_190 * (2 + (input[i] > (762+offset1)));
t1_0_190 += (1 - rt1_0_190) * (input[i] > (760+offset1));

int rt1_0_191 = (input[i] > (765+offset1));
int t1_0_191 = rt1_0_191 * (2 + (input[i] > (766+offset1)));
t1_0_191 += (1 - rt1_0_191) * (input[i] > (764+offset1));

int rt1_0_192 = (input[i] > (769+offset1));
int t1_0_192 = rt1_0_192 * (2 + (input[i] > (770+offset1)));
t1_0_192 += (1 - rt1_0_192) * (input[i] > (768+offset1));

int rt1_0_193 = (input[i] > (773+offset1));
int t1_0_193 = rt1_0_193 * (2 + (input[i] > (774+offset1)));
t1_0_193 += (1 - rt1_0_193) * (input[i] > (772+offset1));

int rt1_0_194 = (input[i] > (777+offset1));
int t1_0_194 = rt1_0_194 * (2 + (input[i] > (778+offset1)));
t1_0_194 += (1 - rt1_0_194) * (input[i] > (776+offset1));

int rt1_0_195 = (input[i] > (781+offset1));
int t1_0_195 = rt1_0_195 * (2 + (input[i] > (782+offset1)));
t1_0_195 += (1 - rt1_0_195) * (input[i] > (780+offset1));

int rt1_0_196 = (input[i] > (785+offset1));
int t1_0_196 = rt1_0_196 * (2 + (input[i] > (786+offset1)));
t1_0_196 += (1 - rt1_0_196) * (input[i] > (784+offset1));

int rt1_0_197 = (input[i] > (789+offset1));
int t1_0_197 = rt1_0_197 * (2 + (input[i] > (790+offset1)));
t1_0_197 += (1 - rt1_0_197) * (input[i] > (788+offset1));

int rt1_0_198 = (input[i] > (793+offset1));
int t1_0_198 = rt1_0_198 * (2 + (input[i] > (794+offset1)));
t1_0_198 += (1 - rt1_0_198) * (input[i] > (792+offset1));

int rt1_0_199 = (input[i] > (797+offset1));
int t1_0_199 = rt1_0_199 * (2 + (input[i] > (798+offset1)));
t1_0_199 += (1 - rt1_0_199) * (input[i] > (796+offset1));

int rt1_0_200 = (input[i] > (801+offset1));
int t1_0_200 = rt1_0_200 * (2 + (input[i] > (802+offset1)));
t1_0_200 += (1 - rt1_0_200) * (input[i] > (800+offset1));

int rt1_0_201 = (input[i] > (805+offset1));
int t1_0_201 = rt1_0_201 * (2 + (input[i] > (806+offset1)));
t1_0_201 += (1 - rt1_0_201) * (input[i] > (804+offset1));

int rt1_0_202 = (input[i] > (809+offset1));
int t1_0_202 = rt1_0_202 * (2 + (input[i] > (810+offset1)));
t1_0_202 += (1 - rt1_0_202) * (input[i] > (808+offset1));

int rt1_0_203 = (input[i] > (813+offset1));
int t1_0_203 = rt1_0_203 * (2 + (input[i] > (814+offset1)));
t1_0_203 += (1 - rt1_0_203) * (input[i] > (812+offset1));

int rt1_0_204 = (input[i] > (817+offset1));
int t1_0_204 = rt1_0_204 * (2 + (input[i] > (818+offset1)));
t1_0_204 += (1 - rt1_0_204) * (input[i] > (816+offset1));

int rt1_0_205 = (input[i] > (821+offset1));
int t1_0_205 = rt1_0_205 * (2 + (input[i] > (822+offset1)));
t1_0_205 += (1 - rt1_0_205) * (input[i] > (820+offset1));

int rt1_0_206 = (input[i] > (825+offset1));
int t1_0_206 = rt1_0_206 * (2 + (input[i] > (826+offset1)));
t1_0_206 += (1 - rt1_0_206) * (input[i] > (824+offset1));

int rt1_0_207 = (input[i] > (829+offset1));
int t1_0_207 = rt1_0_207 * (2 + (input[i] > (830+offset1)));
t1_0_207 += (1 - rt1_0_207) * (input[i] > (828+offset1));

int rt1_0_208 = (input[i] > (833+offset1));
int t1_0_208 = rt1_0_208 * (2 + (input[i] > (834+offset1)));
t1_0_208 += (1 - rt1_0_208) * (input[i] > (832+offset1));

int rt1_0_209 = (input[i] > (837+offset1));
int t1_0_209 = rt1_0_209 * (2 + (input[i] > (838+offset1)));
t1_0_209 += (1 - rt1_0_209) * (input[i] > (836+offset1));

int rt1_0_210 = (input[i] > (841+offset1));
int t1_0_210 = rt1_0_210 * (2 + (input[i] > (842+offset1)));
t1_0_210 += (1 - rt1_0_210) * (input[i] > (840+offset1));

int rt1_0_211 = (input[i] > (845+offset1));
int t1_0_211 = rt1_0_211 * (2 + (input[i] > (846+offset1)));
t1_0_211 += (1 - rt1_0_211) * (input[i] > (844+offset1));

int rt1_0_212 = (input[i] > (849+offset1));
int t1_0_212 = rt1_0_212 * (2 + (input[i] > (850+offset1)));
t1_0_212 += (1 - rt1_0_212) * (input[i] > (848+offset1));

int rt1_0_213 = (input[i] > (853+offset1));
int t1_0_213 = rt1_0_213 * (2 + (input[i] > (854+offset1)));
t1_0_213 += (1 - rt1_0_213) * (input[i] > (852+offset1));

int rt1_0_214 = (input[i] > (857+offset1));
int t1_0_214 = rt1_0_214 * (2 + (input[i] > (858+offset1)));
t1_0_214 += (1 - rt1_0_214) * (input[i] > (856+offset1));

int rt1_0_215 = (input[i] > (861+offset1));
int t1_0_215 = rt1_0_215 * (2 + (input[i] > (862+offset1)));
t1_0_215 += (1 - rt1_0_215) * (input[i] > (860+offset1));

int rt1_0_216 = (input[i] > (865+offset1));
int t1_0_216 = rt1_0_216 * (2 + (input[i] > (866+offset1)));
t1_0_216 += (1 - rt1_0_216) * (input[i] > (864+offset1));

int rt1_0_217 = (input[i] > (869+offset1));
int t1_0_217 = rt1_0_217 * (2 + (input[i] > (870+offset1)));
t1_0_217 += (1 - rt1_0_217) * (input[i] > (868+offset1));

int rt1_0_218 = (input[i] > (873+offset1));
int t1_0_218 = rt1_0_218 * (2 + (input[i] > (874+offset1)));
t1_0_218 += (1 - rt1_0_218) * (input[i] > (872+offset1));

int rt1_0_219 = (input[i] > (877+offset1));
int t1_0_219 = rt1_0_219 * (2 + (input[i] > (878+offset1)));
t1_0_219 += (1 - rt1_0_219) * (input[i] > (876+offset1));

int rt1_0_220 = (input[i] > (881+offset1));
int t1_0_220 = rt1_0_220 * (2 + (input[i] > (882+offset1)));
t1_0_220 += (1 - rt1_0_220) * (input[i] > (880+offset1));

int rt1_0_221 = (input[i] > (885+offset1));
int t1_0_221 = rt1_0_221 * (2 + (input[i] > (886+offset1)));
t1_0_221 += (1 - rt1_0_221) * (input[i] > (884+offset1));

int rt1_0_222 = (input[i] > (889+offset1));
int t1_0_222 = rt1_0_222 * (2 + (input[i] > (890+offset1)));
t1_0_222 += (1 - rt1_0_222) * (input[i] > (888+offset1));

int rt1_0_223 = (input[i] > (893+offset1));
int t1_0_223 = rt1_0_223 * (2 + (input[i] > (894+offset1)));
t1_0_223 += (1 - rt1_0_223) * (input[i] > (892+offset1));

int rt1_0_224 = (input[i] > (897+offset1));
int t1_0_224 = rt1_0_224 * (2 + (input[i] > (898+offset1)));
t1_0_224 += (1 - rt1_0_224) * (input[i] > (896+offset1));

int rt1_0_225 = (input[i] > (901+offset1));
int t1_0_225 = rt1_0_225 * (2 + (input[i] > (902+offset1)));
t1_0_225 += (1 - rt1_0_225) * (input[i] > (900+offset1));

int rt1_0_226 = (input[i] > (905+offset1));
int t1_0_226 = rt1_0_226 * (2 + (input[i] > (906+offset1)));
t1_0_226 += (1 - rt1_0_226) * (input[i] > (904+offset1));

int rt1_0_227 = (input[i] > (909+offset1));
int t1_0_227 = rt1_0_227 * (2 + (input[i] > (910+offset1)));
t1_0_227 += (1 - rt1_0_227) * (input[i] > (908+offset1));

int rt1_0_228 = (input[i] > (913+offset1));
int t1_0_228 = rt1_0_228 * (2 + (input[i] > (914+offset1)));
t1_0_228 += (1 - rt1_0_228) * (input[i] > (912+offset1));

int rt1_0_229 = (input[i] > (917+offset1));
int t1_0_229 = rt1_0_229 * (2 + (input[i] > (918+offset1)));
t1_0_229 += (1 - rt1_0_229) * (input[i] > (916+offset1));

int rt1_0_230 = (input[i] > (921+offset1));
int t1_0_230 = rt1_0_230 * (2 + (input[i] > (922+offset1)));
t1_0_230 += (1 - rt1_0_230) * (input[i] > (920+offset1));

int rt1_0_231 = (input[i] > (925+offset1));
int t1_0_231 = rt1_0_231 * (2 + (input[i] > (926+offset1)));
t1_0_231 += (1 - rt1_0_231) * (input[i] > (924+offset1));

int rt1_0_232 = (input[i] > (929+offset1));
int t1_0_232 = rt1_0_232 * (2 + (input[i] > (930+offset1)));
t1_0_232 += (1 - rt1_0_232) * (input[i] > (928+offset1));

int rt1_0_233 = (input[i] > (933+offset1));
int t1_0_233 = rt1_0_233 * (2 + (input[i] > (934+offset1)));
t1_0_233 += (1 - rt1_0_233) * (input[i] > (932+offset1));

int rt1_0_234 = (input[i] > (937+offset1));
int t1_0_234 = rt1_0_234 * (2 + (input[i] > (938+offset1)));
t1_0_234 += (1 - rt1_0_234) * (input[i] > (936+offset1));

int rt1_0_235 = (input[i] > (941+offset1));
int t1_0_235 = rt1_0_235 * (2 + (input[i] > (942+offset1)));
t1_0_235 += (1 - rt1_0_235) * (input[i] > (940+offset1));

int rt1_0_236 = (input[i] > (945+offset1));
int t1_0_236 = rt1_0_236 * (2 + (input[i] > (946+offset1)));
t1_0_236 += (1 - rt1_0_236) * (input[i] > (944+offset1));

int rt1_0_237 = (input[i] > (949+offset1));
int t1_0_237 = rt1_0_237 * (2 + (input[i] > (950+offset1)));
t1_0_237 += (1 - rt1_0_237) * (input[i] > (948+offset1));

int rt1_0_238 = (input[i] > (953+offset1));
int t1_0_238 = rt1_0_238 * (2 + (input[i] > (954+offset1)));
t1_0_238 += (1 - rt1_0_238) * (input[i] > (952+offset1));

int rt1_0_239 = (input[i] > (957+offset1));
int t1_0_239 = rt1_0_239 * (2 + (input[i] > (958+offset1)));
t1_0_239 += (1 - rt1_0_239) * (input[i] > (956+offset1));

int rt1_0_240 = (input[i] > (961+offset1));
int t1_0_240 = rt1_0_240 * (2 + (input[i] > (962+offset1)));
t1_0_240 += (1 - rt1_0_240) * (input[i] > (960+offset1));

int rt1_0_241 = (input[i] > (965+offset1));
int t1_0_241 = rt1_0_241 * (2 + (input[i] > (966+offset1)));
t1_0_241 += (1 - rt1_0_241) * (input[i] > (964+offset1));

int rt1_0_242 = (input[i] > (969+offset1));
int t1_0_242 = rt1_0_242 * (2 + (input[i] > (970+offset1)));
t1_0_242 += (1 - rt1_0_242) * (input[i] > (968+offset1));

int rt1_0_243 = (input[i] > (973+offset1));
int t1_0_243 = rt1_0_243 * (2 + (input[i] > (974+offset1)));
t1_0_243 += (1 - rt1_0_243) * (input[i] > (972+offset1));

int rt1_0_244 = (input[i] > (977+offset1));
int t1_0_244 = rt1_0_244 * (2 + (input[i] > (978+offset1)));
t1_0_244 += (1 - rt1_0_244) * (input[i] > (976+offset1));

int rt1_0_245 = (input[i] > (981+offset1));
int t1_0_245 = rt1_0_245 * (2 + (input[i] > (982+offset1)));
t1_0_245 += (1 - rt1_0_245) * (input[i] > (980+offset1));

int rt1_0_246 = (input[i] > (985+offset1));
int t1_0_246 = rt1_0_246 * (2 + (input[i] > (986+offset1)));
t1_0_246 += (1 - rt1_0_246) * (input[i] > (984+offset1));

int rt1_0_247 = (input[i] > (989+offset1));
int t1_0_247 = rt1_0_247 * (2 + (input[i] > (990+offset1)));
t1_0_247 += (1 - rt1_0_247) * (input[i] > (988+offset1));

int rt1_0_248 = (input[i] > (993+offset1));
int t1_0_248 = rt1_0_248 * (2 + (input[i] > (994+offset1)));
t1_0_248 += (1 - rt1_0_248) * (input[i] > (992+offset1));

int rt1_0_249 = (input[i] > (997+offset1));
int t1_0_249 = rt1_0_249 * (2 + (input[i] > (998+offset1)));
t1_0_249 += (1 - rt1_0_249) * (input[i] > (996+offset1));

int rt1_0_250 = (input[i] > (1001+offset1));
int t1_0_250 = rt1_0_250 * (2 + (input[i] > (1002+offset1)));
t1_0_250 += (1 - rt1_0_250) * (input[i] > (1000+offset1));

int rt1_0_251 = (input[i] > (1005+offset1));
int t1_0_251 = rt1_0_251 * (2 + (input[i] > (1006+offset1)));
t1_0_251 += (1 - rt1_0_251) * (input[i] > (1004+offset1));

int rt1_0_252 = (input[i] > (1009+offset1));
int t1_0_252 = rt1_0_252 * (2 + (input[i] > (1010+offset1)));
t1_0_252 += (1 - rt1_0_252) * (input[i] > (1008+offset1));

int rt1_0_253 = (input[i] > (1013+offset1));
int t1_0_253 = rt1_0_253 * (2 + (input[i] > (1014+offset1)));
t1_0_253 += (1 - rt1_0_253) * (input[i] > (1012+offset1));

int rt1_0_254 = (input[i] > (1017+offset1));
int t1_0_254 = rt1_0_254 * (2 + (input[i] > (1018+offset1)));
t1_0_254 += (1 - rt1_0_254) * (input[i] > (1016+offset1));

int rt1_0_255 = (input[i] > (1021+offset1));
int t1_0_255 = rt1_0_255 * (2 + (input[i] > (1022+offset1)));
t1_0_255 += (1 - rt1_0_255) * (input[i] > (1020+offset1));

int rt1_1_0 = (input[i] > (3+offset1));
int t1_1_0 = rt1_1_0 * (4 + t1_0_1);
t1_1_0 += (1 - rt1_1_0) * t1_0_0;

int rt1_1_1 = (input[i] > (11+offset1));
int t1_1_1 = rt1_1_1 * (4 + t1_0_3);
t1_1_1 += (1 - rt1_1_1) * t1_0_2;

int rt1_1_2 = (input[i] > (19+offset1));
int t1_1_2 = rt1_1_2 * (4 + t1_0_5);
t1_1_2 += (1 - rt1_1_2) * t1_0_4;

int rt1_1_3 = (input[i] > (27+offset1));
int t1_1_3 = rt1_1_3 * (4 + t1_0_7);
t1_1_3 += (1 - rt1_1_3) * t1_0_6;

int rt1_1_4 = (input[i] > (35+offset1));
int t1_1_4 = rt1_1_4 * (4 + t1_0_9);
t1_1_4 += (1 - rt1_1_4) * t1_0_8;

int rt1_1_5 = (input[i] > (43+offset1));
int t1_1_5 = rt1_1_5 * (4 + t1_0_11);
t1_1_5 += (1 - rt1_1_5) * t1_0_10;

int rt1_1_6 = (input[i] > (51+offset1));
int t1_1_6 = rt1_1_6 * (4 + t1_0_13);
t1_1_6 += (1 - rt1_1_6) * t1_0_12;

int rt1_1_7 = (input[i] > (59+offset1));
int t1_1_7 = rt1_1_7 * (4 + t1_0_15);
t1_1_7 += (1 - rt1_1_7) * t1_0_14;

int rt1_1_8 = (input[i] > (67+offset1));
int t1_1_8 = rt1_1_8 * (4 + t1_0_17);
t1_1_8 += (1 - rt1_1_8) * t1_0_16;

int rt1_1_9 = (input[i] > (75+offset1));
int t1_1_9 = rt1_1_9 * (4 + t1_0_19);
t1_1_9 += (1 - rt1_1_9) * t1_0_18;

int rt1_1_10 = (input[i] > (83+offset1));
int t1_1_10 = rt1_1_10 * (4 + t1_0_21);
t1_1_10 += (1 - rt1_1_10) * t1_0_20;

int rt1_1_11 = (input[i] > (91+offset1));
int t1_1_11 = rt1_1_11 * (4 + t1_0_23);
t1_1_11 += (1 - rt1_1_11) * t1_0_22;

int rt1_1_12 = (input[i] > (99+offset1));
int t1_1_12 = rt1_1_12 * (4 + t1_0_25);
t1_1_12 += (1 - rt1_1_12) * t1_0_24;

int rt1_1_13 = (input[i] > (107+offset1));
int t1_1_13 = rt1_1_13 * (4 + t1_0_27);
t1_1_13 += (1 - rt1_1_13) * t1_0_26;

int rt1_1_14 = (input[i] > (115+offset1));
int t1_1_14 = rt1_1_14 * (4 + t1_0_29);
t1_1_14 += (1 - rt1_1_14) * t1_0_28;

int rt1_1_15 = (input[i] > (123+offset1));
int t1_1_15 = rt1_1_15 * (4 + t1_0_31);
t1_1_15 += (1 - rt1_1_15) * t1_0_30;

int rt1_1_16 = (input[i] > (131+offset1));
int t1_1_16 = rt1_1_16 * (4 + t1_0_33);
t1_1_16 += (1 - rt1_1_16) * t1_0_32;

int rt1_1_17 = (input[i] > (139+offset1));
int t1_1_17 = rt1_1_17 * (4 + t1_0_35);
t1_1_17 += (1 - rt1_1_17) * t1_0_34;

int rt1_1_18 = (input[i] > (147+offset1));
int t1_1_18 = rt1_1_18 * (4 + t1_0_37);
t1_1_18 += (1 - rt1_1_18) * t1_0_36;

int rt1_1_19 = (input[i] > (155+offset1));
int t1_1_19 = rt1_1_19 * (4 + t1_0_39);
t1_1_19 += (1 - rt1_1_19) * t1_0_38;

int rt1_1_20 = (input[i] > (163+offset1));
int t1_1_20 = rt1_1_20 * (4 + t1_0_41);
t1_1_20 += (1 - rt1_1_20) * t1_0_40;

int rt1_1_21 = (input[i] > (171+offset1));
int t1_1_21 = rt1_1_21 * (4 + t1_0_43);
t1_1_21 += (1 - rt1_1_21) * t1_0_42;

int rt1_1_22 = (input[i] > (179+offset1));
int t1_1_22 = rt1_1_22 * (4 + t1_0_45);
t1_1_22 += (1 - rt1_1_22) * t1_0_44;

int rt1_1_23 = (input[i] > (187+offset1));
int t1_1_23 = rt1_1_23 * (4 + t1_0_47);
t1_1_23 += (1 - rt1_1_23) * t1_0_46;

int rt1_1_24 = (input[i] > (195+offset1));
int t1_1_24 = rt1_1_24 * (4 + t1_0_49);
t1_1_24 += (1 - rt1_1_24) * t1_0_48;

int rt1_1_25 = (input[i] > (203+offset1));
int t1_1_25 = rt1_1_25 * (4 + t1_0_51);
t1_1_25 += (1 - rt1_1_25) * t1_0_50;

int rt1_1_26 = (input[i] > (211+offset1));
int t1_1_26 = rt1_1_26 * (4 + t1_0_53);
t1_1_26 += (1 - rt1_1_26) * t1_0_52;

int rt1_1_27 = (input[i] > (219+offset1));
int t1_1_27 = rt1_1_27 * (4 + t1_0_55);
t1_1_27 += (1 - rt1_1_27) * t1_0_54;

int rt1_1_28 = (input[i] > (227+offset1));
int t1_1_28 = rt1_1_28 * (4 + t1_0_57);
t1_1_28 += (1 - rt1_1_28) * t1_0_56;

int rt1_1_29 = (input[i] > (235+offset1));
int t1_1_29 = rt1_1_29 * (4 + t1_0_59);
t1_1_29 += (1 - rt1_1_29) * t1_0_58;

int rt1_1_30 = (input[i] > (243+offset1));
int t1_1_30 = rt1_1_30 * (4 + t1_0_61);
t1_1_30 += (1 - rt1_1_30) * t1_0_60;

int rt1_1_31 = (input[i] > (251+offset1));
int t1_1_31 = rt1_1_31 * (4 + t1_0_63);
t1_1_31 += (1 - rt1_1_31) * t1_0_62;

int rt1_1_32 = (input[i] > (259+offset1));
int t1_1_32 = rt1_1_32 * (4 + t1_0_65);
t1_1_32 += (1 - rt1_1_32) * t1_0_64;

int rt1_1_33 = (input[i] > (267+offset1));
int t1_1_33 = rt1_1_33 * (4 + t1_0_67);
t1_1_33 += (1 - rt1_1_33) * t1_0_66;

int rt1_1_34 = (input[i] > (275+offset1));
int t1_1_34 = rt1_1_34 * (4 + t1_0_69);
t1_1_34 += (1 - rt1_1_34) * t1_0_68;

int rt1_1_35 = (input[i] > (283+offset1));
int t1_1_35 = rt1_1_35 * (4 + t1_0_71);
t1_1_35 += (1 - rt1_1_35) * t1_0_70;

int rt1_1_36 = (input[i] > (291+offset1));
int t1_1_36 = rt1_1_36 * (4 + t1_0_73);
t1_1_36 += (1 - rt1_1_36) * t1_0_72;

int rt1_1_37 = (input[i] > (299+offset1));
int t1_1_37 = rt1_1_37 * (4 + t1_0_75);
t1_1_37 += (1 - rt1_1_37) * t1_0_74;

int rt1_1_38 = (input[i] > (307+offset1));
int t1_1_38 = rt1_1_38 * (4 + t1_0_77);
t1_1_38 += (1 - rt1_1_38) * t1_0_76;

int rt1_1_39 = (input[i] > (315+offset1));
int t1_1_39 = rt1_1_39 * (4 + t1_0_79);
t1_1_39 += (1 - rt1_1_39) * t1_0_78;

int rt1_1_40 = (input[i] > (323+offset1));
int t1_1_40 = rt1_1_40 * (4 + t1_0_81);
t1_1_40 += (1 - rt1_1_40) * t1_0_80;

int rt1_1_41 = (input[i] > (331+offset1));
int t1_1_41 = rt1_1_41 * (4 + t1_0_83);
t1_1_41 += (1 - rt1_1_41) * t1_0_82;

int rt1_1_42 = (input[i] > (339+offset1));
int t1_1_42 = rt1_1_42 * (4 + t1_0_85);
t1_1_42 += (1 - rt1_1_42) * t1_0_84;

int rt1_1_43 = (input[i] > (347+offset1));
int t1_1_43 = rt1_1_43 * (4 + t1_0_87);
t1_1_43 += (1 - rt1_1_43) * t1_0_86;

int rt1_1_44 = (input[i] > (355+offset1));
int t1_1_44 = rt1_1_44 * (4 + t1_0_89);
t1_1_44 += (1 - rt1_1_44) * t1_0_88;

int rt1_1_45 = (input[i] > (363+offset1));
int t1_1_45 = rt1_1_45 * (4 + t1_0_91);
t1_1_45 += (1 - rt1_1_45) * t1_0_90;

int rt1_1_46 = (input[i] > (371+offset1));
int t1_1_46 = rt1_1_46 * (4 + t1_0_93);
t1_1_46 += (1 - rt1_1_46) * t1_0_92;

int rt1_1_47 = (input[i] > (379+offset1));
int t1_1_47 = rt1_1_47 * (4 + t1_0_95);
t1_1_47 += (1 - rt1_1_47) * t1_0_94;

int rt1_1_48 = (input[i] > (387+offset1));
int t1_1_48 = rt1_1_48 * (4 + t1_0_97);
t1_1_48 += (1 - rt1_1_48) * t1_0_96;

int rt1_1_49 = (input[i] > (395+offset1));
int t1_1_49 = rt1_1_49 * (4 + t1_0_99);
t1_1_49 += (1 - rt1_1_49) * t1_0_98;

int rt1_1_50 = (input[i] > (403+offset1));
int t1_1_50 = rt1_1_50 * (4 + t1_0_101);
t1_1_50 += (1 - rt1_1_50) * t1_0_100;

int rt1_1_51 = (input[i] > (411+offset1));
int t1_1_51 = rt1_1_51 * (4 + t1_0_103);
t1_1_51 += (1 - rt1_1_51) * t1_0_102;

int rt1_1_52 = (input[i] > (419+offset1));
int t1_1_52 = rt1_1_52 * (4 + t1_0_105);
t1_1_52 += (1 - rt1_1_52) * t1_0_104;

int rt1_1_53 = (input[i] > (427+offset1));
int t1_1_53 = rt1_1_53 * (4 + t1_0_107);
t1_1_53 += (1 - rt1_1_53) * t1_0_106;

int rt1_1_54 = (input[i] > (435+offset1));
int t1_1_54 = rt1_1_54 * (4 + t1_0_109);
t1_1_54 += (1 - rt1_1_54) * t1_0_108;

int rt1_1_55 = (input[i] > (443+offset1));
int t1_1_55 = rt1_1_55 * (4 + t1_0_111);
t1_1_55 += (1 - rt1_1_55) * t1_0_110;

int rt1_1_56 = (input[i] > (451+offset1));
int t1_1_56 = rt1_1_56 * (4 + t1_0_113);
t1_1_56 += (1 - rt1_1_56) * t1_0_112;

int rt1_1_57 = (input[i] > (459+offset1));
int t1_1_57 = rt1_1_57 * (4 + t1_0_115);
t1_1_57 += (1 - rt1_1_57) * t1_0_114;

int rt1_1_58 = (input[i] > (467+offset1));
int t1_1_58 = rt1_1_58 * (4 + t1_0_117);
t1_1_58 += (1 - rt1_1_58) * t1_0_116;

int rt1_1_59 = (input[i] > (475+offset1));
int t1_1_59 = rt1_1_59 * (4 + t1_0_119);
t1_1_59 += (1 - rt1_1_59) * t1_0_118;

int rt1_1_60 = (input[i] > (483+offset1));
int t1_1_60 = rt1_1_60 * (4 + t1_0_121);
t1_1_60 += (1 - rt1_1_60) * t1_0_120;

int rt1_1_61 = (input[i] > (491+offset1));
int t1_1_61 = rt1_1_61 * (4 + t1_0_123);
t1_1_61 += (1 - rt1_1_61) * t1_0_122;

int rt1_1_62 = (input[i] > (499+offset1));
int t1_1_62 = rt1_1_62 * (4 + t1_0_125);
t1_1_62 += (1 - rt1_1_62) * t1_0_124;

int rt1_1_63 = (input[i] > (507+offset1));
int t1_1_63 = rt1_1_63 * (4 + t1_0_127);
t1_1_63 += (1 - rt1_1_63) * t1_0_126;

int rt1_1_64 = (input[i] > (515+offset1));
int t1_1_64 = rt1_1_64 * (4 + t1_0_129);
t1_1_64 += (1 - rt1_1_64) * t1_0_128;

int rt1_1_65 = (input[i] > (523+offset1));
int t1_1_65 = rt1_1_65 * (4 + t1_0_131);
t1_1_65 += (1 - rt1_1_65) * t1_0_130;

int rt1_1_66 = (input[i] > (531+offset1));
int t1_1_66 = rt1_1_66 * (4 + t1_0_133);
t1_1_66 += (1 - rt1_1_66) * t1_0_132;

int rt1_1_67 = (input[i] > (539+offset1));
int t1_1_67 = rt1_1_67 * (4 + t1_0_135);
t1_1_67 += (1 - rt1_1_67) * t1_0_134;

int rt1_1_68 = (input[i] > (547+offset1));
int t1_1_68 = rt1_1_68 * (4 + t1_0_137);
t1_1_68 += (1 - rt1_1_68) * t1_0_136;

int rt1_1_69 = (input[i] > (555+offset1));
int t1_1_69 = rt1_1_69 * (4 + t1_0_139);
t1_1_69 += (1 - rt1_1_69) * t1_0_138;

int rt1_1_70 = (input[i] > (563+offset1));
int t1_1_70 = rt1_1_70 * (4 + t1_0_141);
t1_1_70 += (1 - rt1_1_70) * t1_0_140;

int rt1_1_71 = (input[i] > (571+offset1));
int t1_1_71 = rt1_1_71 * (4 + t1_0_143);
t1_1_71 += (1 - rt1_1_71) * t1_0_142;

int rt1_1_72 = (input[i] > (579+offset1));
int t1_1_72 = rt1_1_72 * (4 + t1_0_145);
t1_1_72 += (1 - rt1_1_72) * t1_0_144;

int rt1_1_73 = (input[i] > (587+offset1));
int t1_1_73 = rt1_1_73 * (4 + t1_0_147);
t1_1_73 += (1 - rt1_1_73) * t1_0_146;

int rt1_1_74 = (input[i] > (595+offset1));
int t1_1_74 = rt1_1_74 * (4 + t1_0_149);
t1_1_74 += (1 - rt1_1_74) * t1_0_148;

int rt1_1_75 = (input[i] > (603+offset1));
int t1_1_75 = rt1_1_75 * (4 + t1_0_151);
t1_1_75 += (1 - rt1_1_75) * t1_0_150;

int rt1_1_76 = (input[i] > (611+offset1));
int t1_1_76 = rt1_1_76 * (4 + t1_0_153);
t1_1_76 += (1 - rt1_1_76) * t1_0_152;

int rt1_1_77 = (input[i] > (619+offset1));
int t1_1_77 = rt1_1_77 * (4 + t1_0_155);
t1_1_77 += (1 - rt1_1_77) * t1_0_154;

int rt1_1_78 = (input[i] > (627+offset1));
int t1_1_78 = rt1_1_78 * (4 + t1_0_157);
t1_1_78 += (1 - rt1_1_78) * t1_0_156;

int rt1_1_79 = (input[i] > (635+offset1));
int t1_1_79 = rt1_1_79 * (4 + t1_0_159);
t1_1_79 += (1 - rt1_1_79) * t1_0_158;

int rt1_1_80 = (input[i] > (643+offset1));
int t1_1_80 = rt1_1_80 * (4 + t1_0_161);
t1_1_80 += (1 - rt1_1_80) * t1_0_160;

int rt1_1_81 = (input[i] > (651+offset1));
int t1_1_81 = rt1_1_81 * (4 + t1_0_163);
t1_1_81 += (1 - rt1_1_81) * t1_0_162;

int rt1_1_82 = (input[i] > (659+offset1));
int t1_1_82 = rt1_1_82 * (4 + t1_0_165);
t1_1_82 += (1 - rt1_1_82) * t1_0_164;

int rt1_1_83 = (input[i] > (667+offset1));
int t1_1_83 = rt1_1_83 * (4 + t1_0_167);
t1_1_83 += (1 - rt1_1_83) * t1_0_166;

int rt1_1_84 = (input[i] > (675+offset1));
int t1_1_84 = rt1_1_84 * (4 + t1_0_169);
t1_1_84 += (1 - rt1_1_84) * t1_0_168;

int rt1_1_85 = (input[i] > (683+offset1));
int t1_1_85 = rt1_1_85 * (4 + t1_0_171);
t1_1_85 += (1 - rt1_1_85) * t1_0_170;

int rt1_1_86 = (input[i] > (691+offset1));
int t1_1_86 = rt1_1_86 * (4 + t1_0_173);
t1_1_86 += (1 - rt1_1_86) * t1_0_172;

int rt1_1_87 = (input[i] > (699+offset1));
int t1_1_87 = rt1_1_87 * (4 + t1_0_175);
t1_1_87 += (1 - rt1_1_87) * t1_0_174;

int rt1_1_88 = (input[i] > (707+offset1));
int t1_1_88 = rt1_1_88 * (4 + t1_0_177);
t1_1_88 += (1 - rt1_1_88) * t1_0_176;

int rt1_1_89 = (input[i] > (715+offset1));
int t1_1_89 = rt1_1_89 * (4 + t1_0_179);
t1_1_89 += (1 - rt1_1_89) * t1_0_178;

int rt1_1_90 = (input[i] > (723+offset1));
int t1_1_90 = rt1_1_90 * (4 + t1_0_181);
t1_1_90 += (1 - rt1_1_90) * t1_0_180;

int rt1_1_91 = (input[i] > (731+offset1));
int t1_1_91 = rt1_1_91 * (4 + t1_0_183);
t1_1_91 += (1 - rt1_1_91) * t1_0_182;

int rt1_1_92 = (input[i] > (739+offset1));
int t1_1_92 = rt1_1_92 * (4 + t1_0_185);
t1_1_92 += (1 - rt1_1_92) * t1_0_184;

int rt1_1_93 = (input[i] > (747+offset1));
int t1_1_93 = rt1_1_93 * (4 + t1_0_187);
t1_1_93 += (1 - rt1_1_93) * t1_0_186;

int rt1_1_94 = (input[i] > (755+offset1));
int t1_1_94 = rt1_1_94 * (4 + t1_0_189);
t1_1_94 += (1 - rt1_1_94) * t1_0_188;

int rt1_1_95 = (input[i] > (763+offset1));
int t1_1_95 = rt1_1_95 * (4 + t1_0_191);
t1_1_95 += (1 - rt1_1_95) * t1_0_190;

int rt1_1_96 = (input[i] > (771+offset1));
int t1_1_96 = rt1_1_96 * (4 + t1_0_193);
t1_1_96 += (1 - rt1_1_96) * t1_0_192;

int rt1_1_97 = (input[i] > (779+offset1));
int t1_1_97 = rt1_1_97 * (4 + t1_0_195);
t1_1_97 += (1 - rt1_1_97) * t1_0_194;

int rt1_1_98 = (input[i] > (787+offset1));
int t1_1_98 = rt1_1_98 * (4 + t1_0_197);
t1_1_98 += (1 - rt1_1_98) * t1_0_196;

int rt1_1_99 = (input[i] > (795+offset1));
int t1_1_99 = rt1_1_99 * (4 + t1_0_199);
t1_1_99 += (1 - rt1_1_99) * t1_0_198;

int rt1_1_100 = (input[i] > (803+offset1));
int t1_1_100 = rt1_1_100 * (4 + t1_0_201);
t1_1_100 += (1 - rt1_1_100) * t1_0_200;

int rt1_1_101 = (input[i] > (811+offset1));
int t1_1_101 = rt1_1_101 * (4 + t1_0_203);
t1_1_101 += (1 - rt1_1_101) * t1_0_202;

int rt1_1_102 = (input[i] > (819+offset1));
int t1_1_102 = rt1_1_102 * (4 + t1_0_205);
t1_1_102 += (1 - rt1_1_102) * t1_0_204;

int rt1_1_103 = (input[i] > (827+offset1));
int t1_1_103 = rt1_1_103 * (4 + t1_0_207);
t1_1_103 += (1 - rt1_1_103) * t1_0_206;

int rt1_1_104 = (input[i] > (835+offset1));
int t1_1_104 = rt1_1_104 * (4 + t1_0_209);
t1_1_104 += (1 - rt1_1_104) * t1_0_208;

int rt1_1_105 = (input[i] > (843+offset1));
int t1_1_105 = rt1_1_105 * (4 + t1_0_211);
t1_1_105 += (1 - rt1_1_105) * t1_0_210;

int rt1_1_106 = (input[i] > (851+offset1));
int t1_1_106 = rt1_1_106 * (4 + t1_0_213);
t1_1_106 += (1 - rt1_1_106) * t1_0_212;

int rt1_1_107 = (input[i] > (859+offset1));
int t1_1_107 = rt1_1_107 * (4 + t1_0_215);
t1_1_107 += (1 - rt1_1_107) * t1_0_214;

int rt1_1_108 = (input[i] > (867+offset1));
int t1_1_108 = rt1_1_108 * (4 + t1_0_217);
t1_1_108 += (1 - rt1_1_108) * t1_0_216;

int rt1_1_109 = (input[i] > (875+offset1));
int t1_1_109 = rt1_1_109 * (4 + t1_0_219);
t1_1_109 += (1 - rt1_1_109) * t1_0_218;

int rt1_1_110 = (input[i] > (883+offset1));
int t1_1_110 = rt1_1_110 * (4 + t1_0_221);
t1_1_110 += (1 - rt1_1_110) * t1_0_220;

int rt1_1_111 = (input[i] > (891+offset1));
int t1_1_111 = rt1_1_111 * (4 + t1_0_223);
t1_1_111 += (1 - rt1_1_111) * t1_0_222;

int rt1_1_112 = (input[i] > (899+offset1));
int t1_1_112 = rt1_1_112 * (4 + t1_0_225);
t1_1_112 += (1 - rt1_1_112) * t1_0_224;

int rt1_1_113 = (input[i] > (907+offset1));
int t1_1_113 = rt1_1_113 * (4 + t1_0_227);
t1_1_113 += (1 - rt1_1_113) * t1_0_226;

int rt1_1_114 = (input[i] > (915+offset1));
int t1_1_114 = rt1_1_114 * (4 + t1_0_229);
t1_1_114 += (1 - rt1_1_114) * t1_0_228;

int rt1_1_115 = (input[i] > (923+offset1));
int t1_1_115 = rt1_1_115 * (4 + t1_0_231);
t1_1_115 += (1 - rt1_1_115) * t1_0_230;

int rt1_1_116 = (input[i] > (931+offset1));
int t1_1_116 = rt1_1_116 * (4 + t1_0_233);
t1_1_116 += (1 - rt1_1_116) * t1_0_232;

int rt1_1_117 = (input[i] > (939+offset1));
int t1_1_117 = rt1_1_117 * (4 + t1_0_235);
t1_1_117 += (1 - rt1_1_117) * t1_0_234;

int rt1_1_118 = (input[i] > (947+offset1));
int t1_1_118 = rt1_1_118 * (4 + t1_0_237);
t1_1_118 += (1 - rt1_1_118) * t1_0_236;

int rt1_1_119 = (input[i] > (955+offset1));
int t1_1_119 = rt1_1_119 * (4 + t1_0_239);
t1_1_119 += (1 - rt1_1_119) * t1_0_238;

int rt1_1_120 = (input[i] > (963+offset1));
int t1_1_120 = rt1_1_120 * (4 + t1_0_241);
t1_1_120 += (1 - rt1_1_120) * t1_0_240;

int rt1_1_121 = (input[i] > (971+offset1));
int t1_1_121 = rt1_1_121 * (4 + t1_0_243);
t1_1_121 += (1 - rt1_1_121) * t1_0_242;

int rt1_1_122 = (input[i] > (979+offset1));
int t1_1_122 = rt1_1_122 * (4 + t1_0_245);
t1_1_122 += (1 - rt1_1_122) * t1_0_244;

int rt1_1_123 = (input[i] > (987+offset1));
int t1_1_123 = rt1_1_123 * (4 + t1_0_247);
t1_1_123 += (1 - rt1_1_123) * t1_0_246;

int rt1_1_124 = (input[i] > (995+offset1));
int t1_1_124 = rt1_1_124 * (4 + t1_0_249);
t1_1_124 += (1 - rt1_1_124) * t1_0_248;

int rt1_1_125 = (input[i] > (1003+offset1));
int t1_1_125 = rt1_1_125 * (4 + t1_0_251);
t1_1_125 += (1 - rt1_1_125) * t1_0_250;

int rt1_1_126 = (input[i] > (1011+offset1));
int t1_1_126 = rt1_1_126 * (4 + t1_0_253);
t1_1_126 += (1 - rt1_1_126) * t1_0_252;

int rt1_1_127 = (input[i] > (1019+offset1));
int t1_1_127 = rt1_1_127 * (4 + t1_0_255);
t1_1_127 += (1 - rt1_1_127) * t1_0_254;

int rt1_2_0 = (input[i] > (7+offset1));
int t1_2_0 = rt1_2_0 * (8 + t1_1_1);
t1_2_0 += (1 - rt1_2_0) * t1_1_0;

int rt1_2_1 = (input[i] > (23+offset1));
int t1_2_1 = rt1_2_1 * (8 + t1_1_3);
t1_2_1 += (1 - rt1_2_1) * t1_1_2;

int rt1_2_2 = (input[i] > (39+offset1));
int t1_2_2 = rt1_2_2 * (8 + t1_1_5);
t1_2_2 += (1 - rt1_2_2) * t1_1_4;

int rt1_2_3 = (input[i] > (55+offset1));
int t1_2_3 = rt1_2_3 * (8 + t1_1_7);
t1_2_3 += (1 - rt1_2_3) * t1_1_6;

int rt1_2_4 = (input[i] > (71+offset1));
int t1_2_4 = rt1_2_4 * (8 + t1_1_9);
t1_2_4 += (1 - rt1_2_4) * t1_1_8;

int rt1_2_5 = (input[i] > (87+offset1));
int t1_2_5 = rt1_2_5 * (8 + t1_1_11);
t1_2_5 += (1 - rt1_2_5) * t1_1_10;

int rt1_2_6 = (input[i] > (103+offset1));
int t1_2_6 = rt1_2_6 * (8 + t1_1_13);
t1_2_6 += (1 - rt1_2_6) * t1_1_12;

int rt1_2_7 = (input[i] > (119+offset1));
int t1_2_7 = rt1_2_7 * (8 + t1_1_15);
t1_2_7 += (1 - rt1_2_7) * t1_1_14;

int rt1_2_8 = (input[i] > (135+offset1));
int t1_2_8 = rt1_2_8 * (8 + t1_1_17);
t1_2_8 += (1 - rt1_2_8) * t1_1_16;

int rt1_2_9 = (input[i] > (151+offset1));
int t1_2_9 = rt1_2_9 * (8 + t1_1_19);
t1_2_9 += (1 - rt1_2_9) * t1_1_18;

int rt1_2_10 = (input[i] > (167+offset1));
int t1_2_10 = rt1_2_10 * (8 + t1_1_21);
t1_2_10 += (1 - rt1_2_10) * t1_1_20;

int rt1_2_11 = (input[i] > (183+offset1));
int t1_2_11 = rt1_2_11 * (8 + t1_1_23);
t1_2_11 += (1 - rt1_2_11) * t1_1_22;

int rt1_2_12 = (input[i] > (199+offset1));
int t1_2_12 = rt1_2_12 * (8 + t1_1_25);
t1_2_12 += (1 - rt1_2_12) * t1_1_24;

int rt1_2_13 = (input[i] > (215+offset1));
int t1_2_13 = rt1_2_13 * (8 + t1_1_27);
t1_2_13 += (1 - rt1_2_13) * t1_1_26;

int rt1_2_14 = (input[i] > (231+offset1));
int t1_2_14 = rt1_2_14 * (8 + t1_1_29);
t1_2_14 += (1 - rt1_2_14) * t1_1_28;

int rt1_2_15 = (input[i] > (247+offset1));
int t1_2_15 = rt1_2_15 * (8 + t1_1_31);
t1_2_15 += (1 - rt1_2_15) * t1_1_30;

int rt1_2_16 = (input[i] > (263+offset1));
int t1_2_16 = rt1_2_16 * (8 + t1_1_33);
t1_2_16 += (1 - rt1_2_16) * t1_1_32;

int rt1_2_17 = (input[i] > (279+offset1));
int t1_2_17 = rt1_2_17 * (8 + t1_1_35);
t1_2_17 += (1 - rt1_2_17) * t1_1_34;

int rt1_2_18 = (input[i] > (295+offset1));
int t1_2_18 = rt1_2_18 * (8 + t1_1_37);
t1_2_18 += (1 - rt1_2_18) * t1_1_36;

int rt1_2_19 = (input[i] > (311+offset1));
int t1_2_19 = rt1_2_19 * (8 + t1_1_39);
t1_2_19 += (1 - rt1_2_19) * t1_1_38;

int rt1_2_20 = (input[i] > (327+offset1));
int t1_2_20 = rt1_2_20 * (8 + t1_1_41);
t1_2_20 += (1 - rt1_2_20) * t1_1_40;

int rt1_2_21 = (input[i] > (343+offset1));
int t1_2_21 = rt1_2_21 * (8 + t1_1_43);
t1_2_21 += (1 - rt1_2_21) * t1_1_42;

int rt1_2_22 = (input[i] > (359+offset1));
int t1_2_22 = rt1_2_22 * (8 + t1_1_45);
t1_2_22 += (1 - rt1_2_22) * t1_1_44;

int rt1_2_23 = (input[i] > (375+offset1));
int t1_2_23 = rt1_2_23 * (8 + t1_1_47);
t1_2_23 += (1 - rt1_2_23) * t1_1_46;

int rt1_2_24 = (input[i] > (391+offset1));
int t1_2_24 = rt1_2_24 * (8 + t1_1_49);
t1_2_24 += (1 - rt1_2_24) * t1_1_48;

int rt1_2_25 = (input[i] > (407+offset1));
int t1_2_25 = rt1_2_25 * (8 + t1_1_51);
t1_2_25 += (1 - rt1_2_25) * t1_1_50;

int rt1_2_26 = (input[i] > (423+offset1));
int t1_2_26 = rt1_2_26 * (8 + t1_1_53);
t1_2_26 += (1 - rt1_2_26) * t1_1_52;

int rt1_2_27 = (input[i] > (439+offset1));
int t1_2_27 = rt1_2_27 * (8 + t1_1_55);
t1_2_27 += (1 - rt1_2_27) * t1_1_54;

int rt1_2_28 = (input[i] > (455+offset1));
int t1_2_28 = rt1_2_28 * (8 + t1_1_57);
t1_2_28 += (1 - rt1_2_28) * t1_1_56;

int rt1_2_29 = (input[i] > (471+offset1));
int t1_2_29 = rt1_2_29 * (8 + t1_1_59);
t1_2_29 += (1 - rt1_2_29) * t1_1_58;

int rt1_2_30 = (input[i] > (487+offset1));
int t1_2_30 = rt1_2_30 * (8 + t1_1_61);
t1_2_30 += (1 - rt1_2_30) * t1_1_60;

int rt1_2_31 = (input[i] > (503+offset1));
int t1_2_31 = rt1_2_31 * (8 + t1_1_63);
t1_2_31 += (1 - rt1_2_31) * t1_1_62;

int rt1_2_32 = (input[i] > (519+offset1));
int t1_2_32 = rt1_2_32 * (8 + t1_1_65);
t1_2_32 += (1 - rt1_2_32) * t1_1_64;

int rt1_2_33 = (input[i] > (535+offset1));
int t1_2_33 = rt1_2_33 * (8 + t1_1_67);
t1_2_33 += (1 - rt1_2_33) * t1_1_66;

int rt1_2_34 = (input[i] > (551+offset1));
int t1_2_34 = rt1_2_34 * (8 + t1_1_69);
t1_2_34 += (1 - rt1_2_34) * t1_1_68;

int rt1_2_35 = (input[i] > (567+offset1));
int t1_2_35 = rt1_2_35 * (8 + t1_1_71);
t1_2_35 += (1 - rt1_2_35) * t1_1_70;

int rt1_2_36 = (input[i] > (583+offset1));
int t1_2_36 = rt1_2_36 * (8 + t1_1_73);
t1_2_36 += (1 - rt1_2_36) * t1_1_72;

int rt1_2_37 = (input[i] > (599+offset1));
int t1_2_37 = rt1_2_37 * (8 + t1_1_75);
t1_2_37 += (1 - rt1_2_37) * t1_1_74;

int rt1_2_38 = (input[i] > (615+offset1));
int t1_2_38 = rt1_2_38 * (8 + t1_1_77);
t1_2_38 += (1 - rt1_2_38) * t1_1_76;

int rt1_2_39 = (input[i] > (631+offset1));
int t1_2_39 = rt1_2_39 * (8 + t1_1_79);
t1_2_39 += (1 - rt1_2_39) * t1_1_78;

int rt1_2_40 = (input[i] > (647+offset1));
int t1_2_40 = rt1_2_40 * (8 + t1_1_81);
t1_2_40 += (1 - rt1_2_40) * t1_1_80;

int rt1_2_41 = (input[i] > (663+offset1));
int t1_2_41 = rt1_2_41 * (8 + t1_1_83);
t1_2_41 += (1 - rt1_2_41) * t1_1_82;

int rt1_2_42 = (input[i] > (679+offset1));
int t1_2_42 = rt1_2_42 * (8 + t1_1_85);
t1_2_42 += (1 - rt1_2_42) * t1_1_84;

int rt1_2_43 = (input[i] > (695+offset1));
int t1_2_43 = rt1_2_43 * (8 + t1_1_87);
t1_2_43 += (1 - rt1_2_43) * t1_1_86;

int rt1_2_44 = (input[i] > (711+offset1));
int t1_2_44 = rt1_2_44 * (8 + t1_1_89);
t1_2_44 += (1 - rt1_2_44) * t1_1_88;

int rt1_2_45 = (input[i] > (727+offset1));
int t1_2_45 = rt1_2_45 * (8 + t1_1_91);
t1_2_45 += (1 - rt1_2_45) * t1_1_90;

int rt1_2_46 = (input[i] > (743+offset1));
int t1_2_46 = rt1_2_46 * (8 + t1_1_93);
t1_2_46 += (1 - rt1_2_46) * t1_1_92;

int rt1_2_47 = (input[i] > (759+offset1));
int t1_2_47 = rt1_2_47 * (8 + t1_1_95);
t1_2_47 += (1 - rt1_2_47) * t1_1_94;

int rt1_2_48 = (input[i] > (775+offset1));
int t1_2_48 = rt1_2_48 * (8 + t1_1_97);
t1_2_48 += (1 - rt1_2_48) * t1_1_96;

int rt1_2_49 = (input[i] > (791+offset1));
int t1_2_49 = rt1_2_49 * (8 + t1_1_99);
t1_2_49 += (1 - rt1_2_49) * t1_1_98;

int rt1_2_50 = (input[i] > (807+offset1));
int t1_2_50 = rt1_2_50 * (8 + t1_1_101);
t1_2_50 += (1 - rt1_2_50) * t1_1_100;

int rt1_2_51 = (input[i] > (823+offset1));
int t1_2_51 = rt1_2_51 * (8 + t1_1_103);
t1_2_51 += (1 - rt1_2_51) * t1_1_102;

int rt1_2_52 = (input[i] > (839+offset1));
int t1_2_52 = rt1_2_52 * (8 + t1_1_105);
t1_2_52 += (1 - rt1_2_52) * t1_1_104;

int rt1_2_53 = (input[i] > (855+offset1));
int t1_2_53 = rt1_2_53 * (8 + t1_1_107);
t1_2_53 += (1 - rt1_2_53) * t1_1_106;

int rt1_2_54 = (input[i] > (871+offset1));
int t1_2_54 = rt1_2_54 * (8 + t1_1_109);
t1_2_54 += (1 - rt1_2_54) * t1_1_108;

int rt1_2_55 = (input[i] > (887+offset1));
int t1_2_55 = rt1_2_55 * (8 + t1_1_111);
t1_2_55 += (1 - rt1_2_55) * t1_1_110;

int rt1_2_56 = (input[i] > (903+offset1));
int t1_2_56 = rt1_2_56 * (8 + t1_1_113);
t1_2_56 += (1 - rt1_2_56) * t1_1_112;

int rt1_2_57 = (input[i] > (919+offset1));
int t1_2_57 = rt1_2_57 * (8 + t1_1_115);
t1_2_57 += (1 - rt1_2_57) * t1_1_114;

int rt1_2_58 = (input[i] > (935+offset1));
int t1_2_58 = rt1_2_58 * (8 + t1_1_117);
t1_2_58 += (1 - rt1_2_58) * t1_1_116;

int rt1_2_59 = (input[i] > (951+offset1));
int t1_2_59 = rt1_2_59 * (8 + t1_1_119);
t1_2_59 += (1 - rt1_2_59) * t1_1_118;

int rt1_2_60 = (input[i] > (967+offset1));
int t1_2_60 = rt1_2_60 * (8 + t1_1_121);
t1_2_60 += (1 - rt1_2_60) * t1_1_120;

int rt1_2_61 = (input[i] > (983+offset1));
int t1_2_61 = rt1_2_61 * (8 + t1_1_123);
t1_2_61 += (1 - rt1_2_61) * t1_1_122;

int rt1_2_62 = (input[i] > (999+offset1));
int t1_2_62 = rt1_2_62 * (8 + t1_1_125);
t1_2_62 += (1 - rt1_2_62) * t1_1_124;

int rt1_2_63 = (input[i] > (1015+offset1));
int t1_2_63 = rt1_2_63 * (8 + t1_1_127);
t1_2_63 += (1 - rt1_2_63) * t1_1_126;

int rt1_3_0 = (input[i] > (15+offset1));
int t1_3_0 = rt1_3_0 * (16 + t1_2_1);
t1_3_0 += (1 - rt1_3_0) * t1_2_0;

int rt1_3_1 = (input[i] > (47+offset1));
int t1_3_1 = rt1_3_1 * (16 + t1_2_3);
t1_3_1 += (1 - rt1_3_1) * t1_2_2;

int rt1_3_2 = (input[i] > (79+offset1));
int t1_3_2 = rt1_3_2 * (16 + t1_2_5);
t1_3_2 += (1 - rt1_3_2) * t1_2_4;

int rt1_3_3 = (input[i] > (111+offset1));
int t1_3_3 = rt1_3_3 * (16 + t1_2_7);
t1_3_3 += (1 - rt1_3_3) * t1_2_6;

int rt1_3_4 = (input[i] > (143+offset1));
int t1_3_4 = rt1_3_4 * (16 + t1_2_9);
t1_3_4 += (1 - rt1_3_4) * t1_2_8;

int rt1_3_5 = (input[i] > (175+offset1));
int t1_3_5 = rt1_3_5 * (16 + t1_2_11);
t1_3_5 += (1 - rt1_3_5) * t1_2_10;

int rt1_3_6 = (input[i] > (207+offset1));
int t1_3_6 = rt1_3_6 * (16 + t1_2_13);
t1_3_6 += (1 - rt1_3_6) * t1_2_12;

int rt1_3_7 = (input[i] > (239+offset1));
int t1_3_7 = rt1_3_7 * (16 + t1_2_15);
t1_3_7 += (1 - rt1_3_7) * t1_2_14;

int rt1_3_8 = (input[i] > (271+offset1));
int t1_3_8 = rt1_3_8 * (16 + t1_2_17);
t1_3_8 += (1 - rt1_3_8) * t1_2_16;

int rt1_3_9 = (input[i] > (303+offset1));
int t1_3_9 = rt1_3_9 * (16 + t1_2_19);
t1_3_9 += (1 - rt1_3_9) * t1_2_18;

int rt1_3_10 = (input[i] > (335+offset1));
int t1_3_10 = rt1_3_10 * (16 + t1_2_21);
t1_3_10 += (1 - rt1_3_10) * t1_2_20;

int rt1_3_11 = (input[i] > (367+offset1));
int t1_3_11 = rt1_3_11 * (16 + t1_2_23);
t1_3_11 += (1 - rt1_3_11) * t1_2_22;

int rt1_3_12 = (input[i] > (399+offset1));
int t1_3_12 = rt1_3_12 * (16 + t1_2_25);
t1_3_12 += (1 - rt1_3_12) * t1_2_24;

int rt1_3_13 = (input[i] > (431+offset1));
int t1_3_13 = rt1_3_13 * (16 + t1_2_27);
t1_3_13 += (1 - rt1_3_13) * t1_2_26;

int rt1_3_14 = (input[i] > (463+offset1));
int t1_3_14 = rt1_3_14 * (16 + t1_2_29);
t1_3_14 += (1 - rt1_3_14) * t1_2_28;

int rt1_3_15 = (input[i] > (495+offset1));
int t1_3_15 = rt1_3_15 * (16 + t1_2_31);
t1_3_15 += (1 - rt1_3_15) * t1_2_30;

int rt1_3_16 = (input[i] > (527+offset1));
int t1_3_16 = rt1_3_16 * (16 + t1_2_33);
t1_3_16 += (1 - rt1_3_16) * t1_2_32;

int rt1_3_17 = (input[i] > (559+offset1));
int t1_3_17 = rt1_3_17 * (16 + t1_2_35);
t1_3_17 += (1 - rt1_3_17) * t1_2_34;

int rt1_3_18 = (input[i] > (591+offset1));
int t1_3_18 = rt1_3_18 * (16 + t1_2_37);
t1_3_18 += (1 - rt1_3_18) * t1_2_36;

int rt1_3_19 = (input[i] > (623+offset1));
int t1_3_19 = rt1_3_19 * (16 + t1_2_39);
t1_3_19 += (1 - rt1_3_19) * t1_2_38;

int rt1_3_20 = (input[i] > (655+offset1));
int t1_3_20 = rt1_3_20 * (16 + t1_2_41);
t1_3_20 += (1 - rt1_3_20) * t1_2_40;

int rt1_3_21 = (input[i] > (687+offset1));
int t1_3_21 = rt1_3_21 * (16 + t1_2_43);
t1_3_21 += (1 - rt1_3_21) * t1_2_42;

int rt1_3_22 = (input[i] > (719+offset1));
int t1_3_22 = rt1_3_22 * (16 + t1_2_45);
t1_3_22 += (1 - rt1_3_22) * t1_2_44;

int rt1_3_23 = (input[i] > (751+offset1));
int t1_3_23 = rt1_3_23 * (16 + t1_2_47);
t1_3_23 += (1 - rt1_3_23) * t1_2_46;

int rt1_3_24 = (input[i] > (783+offset1));
int t1_3_24 = rt1_3_24 * (16 + t1_2_49);
t1_3_24 += (1 - rt1_3_24) * t1_2_48;

int rt1_3_25 = (input[i] > (815+offset1));
int t1_3_25 = rt1_3_25 * (16 + t1_2_51);
t1_3_25 += (1 - rt1_3_25) * t1_2_50;

int rt1_3_26 = (input[i] > (847+offset1));
int t1_3_26 = rt1_3_26 * (16 + t1_2_53);
t1_3_26 += (1 - rt1_3_26) * t1_2_52;

int rt1_3_27 = (input[i] > (879+offset1));
int t1_3_27 = rt1_3_27 * (16 + t1_2_55);
t1_3_27 += (1 - rt1_3_27) * t1_2_54;

int rt1_3_28 = (input[i] > (911+offset1));
int t1_3_28 = rt1_3_28 * (16 + t1_2_57);
t1_3_28 += (1 - rt1_3_28) * t1_2_56;

int rt1_3_29 = (input[i] > (943+offset1));
int t1_3_29 = rt1_3_29 * (16 + t1_2_59);
t1_3_29 += (1 - rt1_3_29) * t1_2_58;

int rt1_3_30 = (input[i] > (975+offset1));
int t1_3_30 = rt1_3_30 * (16 + t1_2_61);
t1_3_30 += (1 - rt1_3_30) * t1_2_60;

int rt1_3_31 = (input[i] > (1007+offset1));
int t1_3_31 = rt1_3_31 * (16 + t1_2_63);
t1_3_31 += (1 - rt1_3_31) * t1_2_62;

int rt1_4_0 = (input[i] > (31+offset1));
int t1_4_0 = rt1_4_0 * (32 + t1_3_1);
t1_4_0 += (1 - rt1_4_0) * t1_3_0;

int rt1_4_1 = (input[i] > (95+offset1));
int t1_4_1 = rt1_4_1 * (32 + t1_3_3);
t1_4_1 += (1 - rt1_4_1) * t1_3_2;

int rt1_4_2 = (input[i] > (159+offset1));
int t1_4_2 = rt1_4_2 * (32 + t1_3_5);
t1_4_2 += (1 - rt1_4_2) * t1_3_4;

int rt1_4_3 = (input[i] > (223+offset1));
int t1_4_3 = rt1_4_3 * (32 + t1_3_7);
t1_4_3 += (1 - rt1_4_3) * t1_3_6;

int rt1_4_4 = (input[i] > (287+offset1));
int t1_4_4 = rt1_4_4 * (32 + t1_3_9);
t1_4_4 += (1 - rt1_4_4) * t1_3_8;

int rt1_4_5 = (input[i] > (351+offset1));
int t1_4_5 = rt1_4_5 * (32 + t1_3_11);
t1_4_5 += (1 - rt1_4_5) * t1_3_10;

int rt1_4_6 = (input[i] > (415+offset1));
int t1_4_6 = rt1_4_6 * (32 + t1_3_13);
t1_4_6 += (1 - rt1_4_6) * t1_3_12;

int rt1_4_7 = (input[i] > (479+offset1));
int t1_4_7 = rt1_4_7 * (32 + t1_3_15);
t1_4_7 += (1 - rt1_4_7) * t1_3_14;

int rt1_4_8 = (input[i] > (543+offset1));
int t1_4_8 = rt1_4_8 * (32 + t1_3_17);
t1_4_8 += (1 - rt1_4_8) * t1_3_16;

int rt1_4_9 = (input[i] > (607+offset1));
int t1_4_9 = rt1_4_9 * (32 + t1_3_19);
t1_4_9 += (1 - rt1_4_9) * t1_3_18;

int rt1_4_10 = (input[i] > (671+offset1));
int t1_4_10 = rt1_4_10 * (32 + t1_3_21);
t1_4_10 += (1 - rt1_4_10) * t1_3_20;

int rt1_4_11 = (input[i] > (735+offset1));
int t1_4_11 = rt1_4_11 * (32 + t1_3_23);
t1_4_11 += (1 - rt1_4_11) * t1_3_22;

int rt1_4_12 = (input[i] > (799+offset1));
int t1_4_12 = rt1_4_12 * (32 + t1_3_25);
t1_4_12 += (1 - rt1_4_12) * t1_3_24;

int rt1_4_13 = (input[i] > (863+offset1));
int t1_4_13 = rt1_4_13 * (32 + t1_3_27);
t1_4_13 += (1 - rt1_4_13) * t1_3_26;

int rt1_4_14 = (input[i] > (927+offset1));
int t1_4_14 = rt1_4_14 * (32 + t1_3_29);
t1_4_14 += (1 - rt1_4_14) * t1_3_28;

int rt1_4_15 = (input[i] > (991+offset1));
int t1_4_15 = rt1_4_15 * (32 + t1_3_31);
t1_4_15 += (1 - rt1_4_15) * t1_3_30;

int rt1_5_0 = (input[i] > (63+offset1));
int t1_5_0 = rt1_5_0 * (64 + t1_4_1);
t1_5_0 += (1 - rt1_5_0) * t1_4_0;

int rt1_5_1 = (input[i] > (191+offset1));
int t1_5_1 = rt1_5_1 * (64 + t1_4_3);
t1_5_1 += (1 - rt1_5_1) * t1_4_2;

int rt1_5_2 = (input[i] > (319+offset1));
int t1_5_2 = rt1_5_2 * (64 + t1_4_5);
t1_5_2 += (1 - rt1_5_2) * t1_4_4;

int rt1_5_3 = (input[i] > (447+offset1));
int t1_5_3 = rt1_5_3 * (64 + t1_4_7);
t1_5_3 += (1 - rt1_5_3) * t1_4_6;

int rt1_5_4 = (input[i] > (575+offset1));
int t1_5_4 = rt1_5_4 * (64 + t1_4_9);
t1_5_4 += (1 - rt1_5_4) * t1_4_8;

int rt1_5_5 = (input[i] > (703+offset1));
int t1_5_5 = rt1_5_5 * (64 + t1_4_11);
t1_5_5 += (1 - rt1_5_5) * t1_4_10;

int rt1_5_6 = (input[i] > (831+offset1));
int t1_5_6 = rt1_5_6 * (64 + t1_4_13);
t1_5_6 += (1 - rt1_5_6) * t1_4_12;

int rt1_5_7 = (input[i] > (959+offset1));
int t1_5_7 = rt1_5_7 * (64 + t1_4_15);
t1_5_7 += (1 - rt1_5_7) * t1_4_14;

int rt1_6_0 = (input[i] > (127+offset1));
int t1_6_0 = rt1_6_0 * (128 + t1_5_1);
t1_6_0 += (1 - rt1_6_0) * t1_5_0;

int rt1_6_1 = (input[i] > (383+offset1));
int t1_6_1 = rt1_6_1 * (128 + t1_5_3);
t1_6_1 += (1 - rt1_6_1) * t1_5_2;

int rt1_6_2 = (input[i] > (639+offset1));
int t1_6_2 = rt1_6_2 * (128 + t1_5_5);
t1_6_2 += (1 - rt1_6_2) * t1_5_4;

int rt1_6_3 = (input[i] > (895+offset1));
int t1_6_3 = rt1_6_3 * (128 + t1_5_7);
t1_6_3 += (1 - rt1_6_3) * t1_5_6;

int rt1_7_0 = (input[i] > (255+offset1));
int t1_7_0 = rt1_7_0 * (256 + t1_6_1);
t1_7_0 += (1 - rt1_7_0) * t1_6_0;

int rt1_7_1 = (input[i] > (767+offset1));
int t1_7_1 = rt1_7_1 * (256 + t1_6_3);
t1_7_1 += (1 - rt1_7_1) * t1_6_2;

int rt1_8_0 = (input[i] > (511+offset1));
int t1_8_0 = rt1_8_0 * (512 + t1_7_1);
t1_8_0 += (1 - rt1_8_0) * t1_7_0;

int rt2_0_0 = (input[i] > (1+offset2));
int t2_0_0 = rt2_0_0 * (2 + (input[i] > (2+offset2)));
t2_0_0 += (1 - rt2_0_0) * (input[i] > (0+offset2));

int rt2_0_1 = (input[i] > (5+offset2));
int t2_0_1 = rt2_0_1 * (2 + (input[i] > (6+offset2)));
t2_0_1 += (1 - rt2_0_1) * (input[i] > (4+offset2));

int rt2_0_2 = (input[i] > (9+offset2));
int t2_0_2 = rt2_0_2 * (2 + (input[i] > (10+offset2)));
t2_0_2 += (1 - rt2_0_2) * (input[i] > (8+offset2));

int rt2_0_3 = (input[i] > (13+offset2));
int t2_0_3 = rt2_0_3 * (2 + (input[i] > (14+offset2)));
t2_0_3 += (1 - rt2_0_3) * (input[i] > (12+offset2));

int rt2_0_4 = (input[i] > (17+offset2));
int t2_0_4 = rt2_0_4 * (2 + (input[i] > (18+offset2)));
t2_0_4 += (1 - rt2_0_4) * (input[i] > (16+offset2));

int rt2_0_5 = (input[i] > (21+offset2));
int t2_0_5 = rt2_0_5 * (2 + (input[i] > (22+offset2)));
t2_0_5 += (1 - rt2_0_5) * (input[i] > (20+offset2));

int rt2_0_6 = (input[i] > (25+offset2));
int t2_0_6 = rt2_0_6 * (2 + (input[i] > (26+offset2)));
t2_0_6 += (1 - rt2_0_6) * (input[i] > (24+offset2));

int rt2_0_7 = (input[i] > (29+offset2));
int t2_0_7 = rt2_0_7 * (2 + (input[i] > (30+offset2)));
t2_0_7 += (1 - rt2_0_7) * (input[i] > (28+offset2));

int rt2_0_8 = (input[i] > (33+offset2));
int t2_0_8 = rt2_0_8 * (2 + (input[i] > (34+offset2)));
t2_0_8 += (1 - rt2_0_8) * (input[i] > (32+offset2));

int rt2_0_9 = (input[i] > (37+offset2));
int t2_0_9 = rt2_0_9 * (2 + (input[i] > (38+offset2)));
t2_0_9 += (1 - rt2_0_9) * (input[i] > (36+offset2));

int rt2_0_10 = (input[i] > (41+offset2));
int t2_0_10 = rt2_0_10 * (2 + (input[i] > (42+offset2)));
t2_0_10 += (1 - rt2_0_10) * (input[i] > (40+offset2));

int rt2_0_11 = (input[i] > (45+offset2));
int t2_0_11 = rt2_0_11 * (2 + (input[i] > (46+offset2)));
t2_0_11 += (1 - rt2_0_11) * (input[i] > (44+offset2));

int rt2_0_12 = (input[i] > (49+offset2));
int t2_0_12 = rt2_0_12 * (2 + (input[i] > (50+offset2)));
t2_0_12 += (1 - rt2_0_12) * (input[i] > (48+offset2));

int rt2_0_13 = (input[i] > (53+offset2));
int t2_0_13 = rt2_0_13 * (2 + (input[i] > (54+offset2)));
t2_0_13 += (1 - rt2_0_13) * (input[i] > (52+offset2));

int rt2_0_14 = (input[i] > (57+offset2));
int t2_0_14 = rt2_0_14 * (2 + (input[i] > (58+offset2)));
t2_0_14 += (1 - rt2_0_14) * (input[i] > (56+offset2));

int rt2_0_15 = (input[i] > (61+offset2));
int t2_0_15 = rt2_0_15 * (2 + (input[i] > (62+offset2)));
t2_0_15 += (1 - rt2_0_15) * (input[i] > (60+offset2));

int rt2_0_16 = (input[i] > (65+offset2));
int t2_0_16 = rt2_0_16 * (2 + (input[i] > (66+offset2)));
t2_0_16 += (1 - rt2_0_16) * (input[i] > (64+offset2));

int rt2_0_17 = (input[i] > (69+offset2));
int t2_0_17 = rt2_0_17 * (2 + (input[i] > (70+offset2)));
t2_0_17 += (1 - rt2_0_17) * (input[i] > (68+offset2));

int rt2_0_18 = (input[i] > (73+offset2));
int t2_0_18 = rt2_0_18 * (2 + (input[i] > (74+offset2)));
t2_0_18 += (1 - rt2_0_18) * (input[i] > (72+offset2));

int rt2_0_19 = (input[i] > (77+offset2));
int t2_0_19 = rt2_0_19 * (2 + (input[i] > (78+offset2)));
t2_0_19 += (1 - rt2_0_19) * (input[i] > (76+offset2));

int rt2_0_20 = (input[i] > (81+offset2));
int t2_0_20 = rt2_0_20 * (2 + (input[i] > (82+offset2)));
t2_0_20 += (1 - rt2_0_20) * (input[i] > (80+offset2));

int rt2_0_21 = (input[i] > (85+offset2));
int t2_0_21 = rt2_0_21 * (2 + (input[i] > (86+offset2)));
t2_0_21 += (1 - rt2_0_21) * (input[i] > (84+offset2));

int rt2_0_22 = (input[i] > (89+offset2));
int t2_0_22 = rt2_0_22 * (2 + (input[i] > (90+offset2)));
t2_0_22 += (1 - rt2_0_22) * (input[i] > (88+offset2));

int rt2_0_23 = (input[i] > (93+offset2));
int t2_0_23 = rt2_0_23 * (2 + (input[i] > (94+offset2)));
t2_0_23 += (1 - rt2_0_23) * (input[i] > (92+offset2));

int rt2_0_24 = (input[i] > (97+offset2));
int t2_0_24 = rt2_0_24 * (2 + (input[i] > (98+offset2)));
t2_0_24 += (1 - rt2_0_24) * (input[i] > (96+offset2));

int rt2_0_25 = (input[i] > (101+offset2));
int t2_0_25 = rt2_0_25 * (2 + (input[i] > (102+offset2)));
t2_0_25 += (1 - rt2_0_25) * (input[i] > (100+offset2));

int rt2_0_26 = (input[i] > (105+offset2));
int t2_0_26 = rt2_0_26 * (2 + (input[i] > (106+offset2)));
t2_0_26 += (1 - rt2_0_26) * (input[i] > (104+offset2));

int rt2_0_27 = (input[i] > (109+offset2));
int t2_0_27 = rt2_0_27 * (2 + (input[i] > (110+offset2)));
t2_0_27 += (1 - rt2_0_27) * (input[i] > (108+offset2));

int rt2_0_28 = (input[i] > (113+offset2));
int t2_0_28 = rt2_0_28 * (2 + (input[i] > (114+offset2)));
t2_0_28 += (1 - rt2_0_28) * (input[i] > (112+offset2));

int rt2_0_29 = (input[i] > (117+offset2));
int t2_0_29 = rt2_0_29 * (2 + (input[i] > (118+offset2)));
t2_0_29 += (1 - rt2_0_29) * (input[i] > (116+offset2));

int rt2_0_30 = (input[i] > (121+offset2));
int t2_0_30 = rt2_0_30 * (2 + (input[i] > (122+offset2)));
t2_0_30 += (1 - rt2_0_30) * (input[i] > (120+offset2));

int rt2_0_31 = (input[i] > (125+offset2));
int t2_0_31 = rt2_0_31 * (2 + (input[i] > (126+offset2)));
t2_0_31 += (1 - rt2_0_31) * (input[i] > (124+offset2));

int rt2_0_32 = (input[i] > (129+offset2));
int t2_0_32 = rt2_0_32 * (2 + (input[i] > (130+offset2)));
t2_0_32 += (1 - rt2_0_32) * (input[i] > (128+offset2));

int rt2_0_33 = (input[i] > (133+offset2));
int t2_0_33 = rt2_0_33 * (2 + (input[i] > (134+offset2)));
t2_0_33 += (1 - rt2_0_33) * (input[i] > (132+offset2));

int rt2_0_34 = (input[i] > (137+offset2));
int t2_0_34 = rt2_0_34 * (2 + (input[i] > (138+offset2)));
t2_0_34 += (1 - rt2_0_34) * (input[i] > (136+offset2));

int rt2_0_35 = (input[i] > (141+offset2));
int t2_0_35 = rt2_0_35 * (2 + (input[i] > (142+offset2)));
t2_0_35 += (1 - rt2_0_35) * (input[i] > (140+offset2));

int rt2_0_36 = (input[i] > (145+offset2));
int t2_0_36 = rt2_0_36 * (2 + (input[i] > (146+offset2)));
t2_0_36 += (1 - rt2_0_36) * (input[i] > (144+offset2));

int rt2_0_37 = (input[i] > (149+offset2));
int t2_0_37 = rt2_0_37 * (2 + (input[i] > (150+offset2)));
t2_0_37 += (1 - rt2_0_37) * (input[i] > (148+offset2));

int rt2_0_38 = (input[i] > (153+offset2));
int t2_0_38 = rt2_0_38 * (2 + (input[i] > (154+offset2)));
t2_0_38 += (1 - rt2_0_38) * (input[i] > (152+offset2));

int rt2_0_39 = (input[i] > (157+offset2));
int t2_0_39 = rt2_0_39 * (2 + (input[i] > (158+offset2)));
t2_0_39 += (1 - rt2_0_39) * (input[i] > (156+offset2));

int rt2_0_40 = (input[i] > (161+offset2));
int t2_0_40 = rt2_0_40 * (2 + (input[i] > (162+offset2)));
t2_0_40 += (1 - rt2_0_40) * (input[i] > (160+offset2));

int rt2_0_41 = (input[i] > (165+offset2));
int t2_0_41 = rt2_0_41 * (2 + (input[i] > (166+offset2)));
t2_0_41 += (1 - rt2_0_41) * (input[i] > (164+offset2));

int rt2_0_42 = (input[i] > (169+offset2));
int t2_0_42 = rt2_0_42 * (2 + (input[i] > (170+offset2)));
t2_0_42 += (1 - rt2_0_42) * (input[i] > (168+offset2));

int rt2_0_43 = (input[i] > (173+offset2));
int t2_0_43 = rt2_0_43 * (2 + (input[i] > (174+offset2)));
t2_0_43 += (1 - rt2_0_43) * (input[i] > (172+offset2));

int rt2_0_44 = (input[i] > (177+offset2));
int t2_0_44 = rt2_0_44 * (2 + (input[i] > (178+offset2)));
t2_0_44 += (1 - rt2_0_44) * (input[i] > (176+offset2));

int rt2_0_45 = (input[i] > (181+offset2));
int t2_0_45 = rt2_0_45 * (2 + (input[i] > (182+offset2)));
t2_0_45 += (1 - rt2_0_45) * (input[i] > (180+offset2));

int rt2_0_46 = (input[i] > (185+offset2));
int t2_0_46 = rt2_0_46 * (2 + (input[i] > (186+offset2)));
t2_0_46 += (1 - rt2_0_46) * (input[i] > (184+offset2));

int rt2_0_47 = (input[i] > (189+offset2));
int t2_0_47 = rt2_0_47 * (2 + (input[i] > (190+offset2)));
t2_0_47 += (1 - rt2_0_47) * (input[i] > (188+offset2));

int rt2_0_48 = (input[i] > (193+offset2));
int t2_0_48 = rt2_0_48 * (2 + (input[i] > (194+offset2)));
t2_0_48 += (1 - rt2_0_48) * (input[i] > (192+offset2));

int rt2_0_49 = (input[i] > (197+offset2));
int t2_0_49 = rt2_0_49 * (2 + (input[i] > (198+offset2)));
t2_0_49 += (1 - rt2_0_49) * (input[i] > (196+offset2));

int rt2_0_50 = (input[i] > (201+offset2));
int t2_0_50 = rt2_0_50 * (2 + (input[i] > (202+offset2)));
t2_0_50 += (1 - rt2_0_50) * (input[i] > (200+offset2));

int rt2_0_51 = (input[i] > (205+offset2));
int t2_0_51 = rt2_0_51 * (2 + (input[i] > (206+offset2)));
t2_0_51 += (1 - rt2_0_51) * (input[i] > (204+offset2));

int rt2_0_52 = (input[i] > (209+offset2));
int t2_0_52 = rt2_0_52 * (2 + (input[i] > (210+offset2)));
t2_0_52 += (1 - rt2_0_52) * (input[i] > (208+offset2));

int rt2_0_53 = (input[i] > (213+offset2));
int t2_0_53 = rt2_0_53 * (2 + (input[i] > (214+offset2)));
t2_0_53 += (1 - rt2_0_53) * (input[i] > (212+offset2));

int rt2_0_54 = (input[i] > (217+offset2));
int t2_0_54 = rt2_0_54 * (2 + (input[i] > (218+offset2)));
t2_0_54 += (1 - rt2_0_54) * (input[i] > (216+offset2));

int rt2_0_55 = (input[i] > (221+offset2));
int t2_0_55 = rt2_0_55 * (2 + (input[i] > (222+offset2)));
t2_0_55 += (1 - rt2_0_55) * (input[i] > (220+offset2));

int rt2_0_56 = (input[i] > (225+offset2));
int t2_0_56 = rt2_0_56 * (2 + (input[i] > (226+offset2)));
t2_0_56 += (1 - rt2_0_56) * (input[i] > (224+offset2));

int rt2_0_57 = (input[i] > (229+offset2));
int t2_0_57 = rt2_0_57 * (2 + (input[i] > (230+offset2)));
t2_0_57 += (1 - rt2_0_57) * (input[i] > (228+offset2));

int rt2_0_58 = (input[i] > (233+offset2));
int t2_0_58 = rt2_0_58 * (2 + (input[i] > (234+offset2)));
t2_0_58 += (1 - rt2_0_58) * (input[i] > (232+offset2));

int rt2_0_59 = (input[i] > (237+offset2));
int t2_0_59 = rt2_0_59 * (2 + (input[i] > (238+offset2)));
t2_0_59 += (1 - rt2_0_59) * (input[i] > (236+offset2));

int rt2_0_60 = (input[i] > (241+offset2));
int t2_0_60 = rt2_0_60 * (2 + (input[i] > (242+offset2)));
t2_0_60 += (1 - rt2_0_60) * (input[i] > (240+offset2));

int rt2_0_61 = (input[i] > (245+offset2));
int t2_0_61 = rt2_0_61 * (2 + (input[i] > (246+offset2)));
t2_0_61 += (1 - rt2_0_61) * (input[i] > (244+offset2));

int rt2_0_62 = (input[i] > (249+offset2));
int t2_0_62 = rt2_0_62 * (2 + (input[i] > (250+offset2)));
t2_0_62 += (1 - rt2_0_62) * (input[i] > (248+offset2));

int rt2_0_63 = (input[i] > (253+offset2));
int t2_0_63 = rt2_0_63 * (2 + (input[i] > (254+offset2)));
t2_0_63 += (1 - rt2_0_63) * (input[i] > (252+offset2));

int rt2_0_64 = (input[i] > (257+offset2));
int t2_0_64 = rt2_0_64 * (2 + (input[i] > (258+offset2)));
t2_0_64 += (1 - rt2_0_64) * (input[i] > (256+offset2));

int rt2_0_65 = (input[i] > (261+offset2));
int t2_0_65 = rt2_0_65 * (2 + (input[i] > (262+offset2)));
t2_0_65 += (1 - rt2_0_65) * (input[i] > (260+offset2));

int rt2_0_66 = (input[i] > (265+offset2));
int t2_0_66 = rt2_0_66 * (2 + (input[i] > (266+offset2)));
t2_0_66 += (1 - rt2_0_66) * (input[i] > (264+offset2));

int rt2_0_67 = (input[i] > (269+offset2));
int t2_0_67 = rt2_0_67 * (2 + (input[i] > (270+offset2)));
t2_0_67 += (1 - rt2_0_67) * (input[i] > (268+offset2));

int rt2_0_68 = (input[i] > (273+offset2));
int t2_0_68 = rt2_0_68 * (2 + (input[i] > (274+offset2)));
t2_0_68 += (1 - rt2_0_68) * (input[i] > (272+offset2));

int rt2_0_69 = (input[i] > (277+offset2));
int t2_0_69 = rt2_0_69 * (2 + (input[i] > (278+offset2)));
t2_0_69 += (1 - rt2_0_69) * (input[i] > (276+offset2));

int rt2_0_70 = (input[i] > (281+offset2));
int t2_0_70 = rt2_0_70 * (2 + (input[i] > (282+offset2)));
t2_0_70 += (1 - rt2_0_70) * (input[i] > (280+offset2));

int rt2_0_71 = (input[i] > (285+offset2));
int t2_0_71 = rt2_0_71 * (2 + (input[i] > (286+offset2)));
t2_0_71 += (1 - rt2_0_71) * (input[i] > (284+offset2));

int rt2_0_72 = (input[i] > (289+offset2));
int t2_0_72 = rt2_0_72 * (2 + (input[i] > (290+offset2)));
t2_0_72 += (1 - rt2_0_72) * (input[i] > (288+offset2));

int rt2_0_73 = (input[i] > (293+offset2));
int t2_0_73 = rt2_0_73 * (2 + (input[i] > (294+offset2)));
t2_0_73 += (1 - rt2_0_73) * (input[i] > (292+offset2));

int rt2_0_74 = (input[i] > (297+offset2));
int t2_0_74 = rt2_0_74 * (2 + (input[i] > (298+offset2)));
t2_0_74 += (1 - rt2_0_74) * (input[i] > (296+offset2));

int rt2_0_75 = (input[i] > (301+offset2));
int t2_0_75 = rt2_0_75 * (2 + (input[i] > (302+offset2)));
t2_0_75 += (1 - rt2_0_75) * (input[i] > (300+offset2));

int rt2_0_76 = (input[i] > (305+offset2));
int t2_0_76 = rt2_0_76 * (2 + (input[i] > (306+offset2)));
t2_0_76 += (1 - rt2_0_76) * (input[i] > (304+offset2));

int rt2_0_77 = (input[i] > (309+offset2));
int t2_0_77 = rt2_0_77 * (2 + (input[i] > (310+offset2)));
t2_0_77 += (1 - rt2_0_77) * (input[i] > (308+offset2));

int rt2_0_78 = (input[i] > (313+offset2));
int t2_0_78 = rt2_0_78 * (2 + (input[i] > (314+offset2)));
t2_0_78 += (1 - rt2_0_78) * (input[i] > (312+offset2));

int rt2_0_79 = (input[i] > (317+offset2));
int t2_0_79 = rt2_0_79 * (2 + (input[i] > (318+offset2)));
t2_0_79 += (1 - rt2_0_79) * (input[i] > (316+offset2));

int rt2_0_80 = (input[i] > (321+offset2));
int t2_0_80 = rt2_0_80 * (2 + (input[i] > (322+offset2)));
t2_0_80 += (1 - rt2_0_80) * (input[i] > (320+offset2));

int rt2_0_81 = (input[i] > (325+offset2));
int t2_0_81 = rt2_0_81 * (2 + (input[i] > (326+offset2)));
t2_0_81 += (1 - rt2_0_81) * (input[i] > (324+offset2));

int rt2_0_82 = (input[i] > (329+offset2));
int t2_0_82 = rt2_0_82 * (2 + (input[i] > (330+offset2)));
t2_0_82 += (1 - rt2_0_82) * (input[i] > (328+offset2));

int rt2_0_83 = (input[i] > (333+offset2));
int t2_0_83 = rt2_0_83 * (2 + (input[i] > (334+offset2)));
t2_0_83 += (1 - rt2_0_83) * (input[i] > (332+offset2));

int rt2_0_84 = (input[i] > (337+offset2));
int t2_0_84 = rt2_0_84 * (2 + (input[i] > (338+offset2)));
t2_0_84 += (1 - rt2_0_84) * (input[i] > (336+offset2));

int rt2_0_85 = (input[i] > (341+offset2));
int t2_0_85 = rt2_0_85 * (2 + (input[i] > (342+offset2)));
t2_0_85 += (1 - rt2_0_85) * (input[i] > (340+offset2));

int rt2_0_86 = (input[i] > (345+offset2));
int t2_0_86 = rt2_0_86 * (2 + (input[i] > (346+offset2)));
t2_0_86 += (1 - rt2_0_86) * (input[i] > (344+offset2));

int rt2_0_87 = (input[i] > (349+offset2));
int t2_0_87 = rt2_0_87 * (2 + (input[i] > (350+offset2)));
t2_0_87 += (1 - rt2_0_87) * (input[i] > (348+offset2));

int rt2_0_88 = (input[i] > (353+offset2));
int t2_0_88 = rt2_0_88 * (2 + (input[i] > (354+offset2)));
t2_0_88 += (1 - rt2_0_88) * (input[i] > (352+offset2));

int rt2_0_89 = (input[i] > (357+offset2));
int t2_0_89 = rt2_0_89 * (2 + (input[i] > (358+offset2)));
t2_0_89 += (1 - rt2_0_89) * (input[i] > (356+offset2));

int rt2_0_90 = (input[i] > (361+offset2));
int t2_0_90 = rt2_0_90 * (2 + (input[i] > (362+offset2)));
t2_0_90 += (1 - rt2_0_90) * (input[i] > (360+offset2));

int rt2_0_91 = (input[i] > (365+offset2));
int t2_0_91 = rt2_0_91 * (2 + (input[i] > (366+offset2)));
t2_0_91 += (1 - rt2_0_91) * (input[i] > (364+offset2));

int rt2_0_92 = (input[i] > (369+offset2));
int t2_0_92 = rt2_0_92 * (2 + (input[i] > (370+offset2)));
t2_0_92 += (1 - rt2_0_92) * (input[i] > (368+offset2));

int rt2_0_93 = (input[i] > (373+offset2));
int t2_0_93 = rt2_0_93 * (2 + (input[i] > (374+offset2)));
t2_0_93 += (1 - rt2_0_93) * (input[i] > (372+offset2));

int rt2_0_94 = (input[i] > (377+offset2));
int t2_0_94 = rt2_0_94 * (2 + (input[i] > (378+offset2)));
t2_0_94 += (1 - rt2_0_94) * (input[i] > (376+offset2));

int rt2_0_95 = (input[i] > (381+offset2));
int t2_0_95 = rt2_0_95 * (2 + (input[i] > (382+offset2)));
t2_0_95 += (1 - rt2_0_95) * (input[i] > (380+offset2));

int rt2_0_96 = (input[i] > (385+offset2));
int t2_0_96 = rt2_0_96 * (2 + (input[i] > (386+offset2)));
t2_0_96 += (1 - rt2_0_96) * (input[i] > (384+offset2));

int rt2_0_97 = (input[i] > (389+offset2));
int t2_0_97 = rt2_0_97 * (2 + (input[i] > (390+offset2)));
t2_0_97 += (1 - rt2_0_97) * (input[i] > (388+offset2));

int rt2_0_98 = (input[i] > (393+offset2));
int t2_0_98 = rt2_0_98 * (2 + (input[i] > (394+offset2)));
t2_0_98 += (1 - rt2_0_98) * (input[i] > (392+offset2));

int rt2_0_99 = (input[i] > (397+offset2));
int t2_0_99 = rt2_0_99 * (2 + (input[i] > (398+offset2)));
t2_0_99 += (1 - rt2_0_99) * (input[i] > (396+offset2));

int rt2_0_100 = (input[i] > (401+offset2));
int t2_0_100 = rt2_0_100 * (2 + (input[i] > (402+offset2)));
t2_0_100 += (1 - rt2_0_100) * (input[i] > (400+offset2));

int rt2_0_101 = (input[i] > (405+offset2));
int t2_0_101 = rt2_0_101 * (2 + (input[i] > (406+offset2)));
t2_0_101 += (1 - rt2_0_101) * (input[i] > (404+offset2));

int rt2_0_102 = (input[i] > (409+offset2));
int t2_0_102 = rt2_0_102 * (2 + (input[i] > (410+offset2)));
t2_0_102 += (1 - rt2_0_102) * (input[i] > (408+offset2));

int rt2_0_103 = (input[i] > (413+offset2));
int t2_0_103 = rt2_0_103 * (2 + (input[i] > (414+offset2)));
t2_0_103 += (1 - rt2_0_103) * (input[i] > (412+offset2));

int rt2_0_104 = (input[i] > (417+offset2));
int t2_0_104 = rt2_0_104 * (2 + (input[i] > (418+offset2)));
t2_0_104 += (1 - rt2_0_104) * (input[i] > (416+offset2));

int rt2_0_105 = (input[i] > (421+offset2));
int t2_0_105 = rt2_0_105 * (2 + (input[i] > (422+offset2)));
t2_0_105 += (1 - rt2_0_105) * (input[i] > (420+offset2));

int rt2_0_106 = (input[i] > (425+offset2));
int t2_0_106 = rt2_0_106 * (2 + (input[i] > (426+offset2)));
t2_0_106 += (1 - rt2_0_106) * (input[i] > (424+offset2));

int rt2_0_107 = (input[i] > (429+offset2));
int t2_0_107 = rt2_0_107 * (2 + (input[i] > (430+offset2)));
t2_0_107 += (1 - rt2_0_107) * (input[i] > (428+offset2));

int rt2_0_108 = (input[i] > (433+offset2));
int t2_0_108 = rt2_0_108 * (2 + (input[i] > (434+offset2)));
t2_0_108 += (1 - rt2_0_108) * (input[i] > (432+offset2));

int rt2_0_109 = (input[i] > (437+offset2));
int t2_0_109 = rt2_0_109 * (2 + (input[i] > (438+offset2)));
t2_0_109 += (1 - rt2_0_109) * (input[i] > (436+offset2));

int rt2_0_110 = (input[i] > (441+offset2));
int t2_0_110 = rt2_0_110 * (2 + (input[i] > (442+offset2)));
t2_0_110 += (1 - rt2_0_110) * (input[i] > (440+offset2));

int rt2_0_111 = (input[i] > (445+offset2));
int t2_0_111 = rt2_0_111 * (2 + (input[i] > (446+offset2)));
t2_0_111 += (1 - rt2_0_111) * (input[i] > (444+offset2));

int rt2_0_112 = (input[i] > (449+offset2));
int t2_0_112 = rt2_0_112 * (2 + (input[i] > (450+offset2)));
t2_0_112 += (1 - rt2_0_112) * (input[i] > (448+offset2));

int rt2_0_113 = (input[i] > (453+offset2));
int t2_0_113 = rt2_0_113 * (2 + (input[i] > (454+offset2)));
t2_0_113 += (1 - rt2_0_113) * (input[i] > (452+offset2));

int rt2_0_114 = (input[i] > (457+offset2));
int t2_0_114 = rt2_0_114 * (2 + (input[i] > (458+offset2)));
t2_0_114 += (1 - rt2_0_114) * (input[i] > (456+offset2));

int rt2_0_115 = (input[i] > (461+offset2));
int t2_0_115 = rt2_0_115 * (2 + (input[i] > (462+offset2)));
t2_0_115 += (1 - rt2_0_115) * (input[i] > (460+offset2));

int rt2_0_116 = (input[i] > (465+offset2));
int t2_0_116 = rt2_0_116 * (2 + (input[i] > (466+offset2)));
t2_0_116 += (1 - rt2_0_116) * (input[i] > (464+offset2));

int rt2_0_117 = (input[i] > (469+offset2));
int t2_0_117 = rt2_0_117 * (2 + (input[i] > (470+offset2)));
t2_0_117 += (1 - rt2_0_117) * (input[i] > (468+offset2));

int rt2_0_118 = (input[i] > (473+offset2));
int t2_0_118 = rt2_0_118 * (2 + (input[i] > (474+offset2)));
t2_0_118 += (1 - rt2_0_118) * (input[i] > (472+offset2));

int rt2_0_119 = (input[i] > (477+offset2));
int t2_0_119 = rt2_0_119 * (2 + (input[i] > (478+offset2)));
t2_0_119 += (1 - rt2_0_119) * (input[i] > (476+offset2));

int rt2_0_120 = (input[i] > (481+offset2));
int t2_0_120 = rt2_0_120 * (2 + (input[i] > (482+offset2)));
t2_0_120 += (1 - rt2_0_120) * (input[i] > (480+offset2));

int rt2_0_121 = (input[i] > (485+offset2));
int t2_0_121 = rt2_0_121 * (2 + (input[i] > (486+offset2)));
t2_0_121 += (1 - rt2_0_121) * (input[i] > (484+offset2));

int rt2_0_122 = (input[i] > (489+offset2));
int t2_0_122 = rt2_0_122 * (2 + (input[i] > (490+offset2)));
t2_0_122 += (1 - rt2_0_122) * (input[i] > (488+offset2));

int rt2_0_123 = (input[i] > (493+offset2));
int t2_0_123 = rt2_0_123 * (2 + (input[i] > (494+offset2)));
t2_0_123 += (1 - rt2_0_123) * (input[i] > (492+offset2));

int rt2_0_124 = (input[i] > (497+offset2));
int t2_0_124 = rt2_0_124 * (2 + (input[i] > (498+offset2)));
t2_0_124 += (1 - rt2_0_124) * (input[i] > (496+offset2));

int rt2_0_125 = (input[i] > (501+offset2));
int t2_0_125 = rt2_0_125 * (2 + (input[i] > (502+offset2)));
t2_0_125 += (1 - rt2_0_125) * (input[i] > (500+offset2));

int rt2_0_126 = (input[i] > (505+offset2));
int t2_0_126 = rt2_0_126 * (2 + (input[i] > (506+offset2)));
t2_0_126 += (1 - rt2_0_126) * (input[i] > (504+offset2));

int rt2_0_127 = (input[i] > (509+offset2));
int t2_0_127 = rt2_0_127 * (2 + (input[i] > (510+offset2)));
t2_0_127 += (1 - rt2_0_127) * (input[i] > (508+offset2));

int rt2_0_128 = (input[i] > (513+offset2));
int t2_0_128 = rt2_0_128 * (2 + (input[i] > (514+offset2)));
t2_0_128 += (1 - rt2_0_128) * (input[i] > (512+offset2));

int rt2_0_129 = (input[i] > (517+offset2));
int t2_0_129 = rt2_0_129 * (2 + (input[i] > (518+offset2)));
t2_0_129 += (1 - rt2_0_129) * (input[i] > (516+offset2));

int rt2_0_130 = (input[i] > (521+offset2));
int t2_0_130 = rt2_0_130 * (2 + (input[i] > (522+offset2)));
t2_0_130 += (1 - rt2_0_130) * (input[i] > (520+offset2));

int rt2_0_131 = (input[i] > (525+offset2));
int t2_0_131 = rt2_0_131 * (2 + (input[i] > (526+offset2)));
t2_0_131 += (1 - rt2_0_131) * (input[i] > (524+offset2));

int rt2_0_132 = (input[i] > (529+offset2));
int t2_0_132 = rt2_0_132 * (2 + (input[i] > (530+offset2)));
t2_0_132 += (1 - rt2_0_132) * (input[i] > (528+offset2));

int rt2_0_133 = (input[i] > (533+offset2));
int t2_0_133 = rt2_0_133 * (2 + (input[i] > (534+offset2)));
t2_0_133 += (1 - rt2_0_133) * (input[i] > (532+offset2));

int rt2_0_134 = (input[i] > (537+offset2));
int t2_0_134 = rt2_0_134 * (2 + (input[i] > (538+offset2)));
t2_0_134 += (1 - rt2_0_134) * (input[i] > (536+offset2));

int rt2_0_135 = (input[i] > (541+offset2));
int t2_0_135 = rt2_0_135 * (2 + (input[i] > (542+offset2)));
t2_0_135 += (1 - rt2_0_135) * (input[i] > (540+offset2));

int rt2_0_136 = (input[i] > (545+offset2));
int t2_0_136 = rt2_0_136 * (2 + (input[i] > (546+offset2)));
t2_0_136 += (1 - rt2_0_136) * (input[i] > (544+offset2));

int rt2_0_137 = (input[i] > (549+offset2));
int t2_0_137 = rt2_0_137 * (2 + (input[i] > (550+offset2)));
t2_0_137 += (1 - rt2_0_137) * (input[i] > (548+offset2));

int rt2_0_138 = (input[i] > (553+offset2));
int t2_0_138 = rt2_0_138 * (2 + (input[i] > (554+offset2)));
t2_0_138 += (1 - rt2_0_138) * (input[i] > (552+offset2));

int rt2_0_139 = (input[i] > (557+offset2));
int t2_0_139 = rt2_0_139 * (2 + (input[i] > (558+offset2)));
t2_0_139 += (1 - rt2_0_139) * (input[i] > (556+offset2));

int rt2_0_140 = (input[i] > (561+offset2));
int t2_0_140 = rt2_0_140 * (2 + (input[i] > (562+offset2)));
t2_0_140 += (1 - rt2_0_140) * (input[i] > (560+offset2));

int rt2_0_141 = (input[i] > (565+offset2));
int t2_0_141 = rt2_0_141 * (2 + (input[i] > (566+offset2)));
t2_0_141 += (1 - rt2_0_141) * (input[i] > (564+offset2));

int rt2_0_142 = (input[i] > (569+offset2));
int t2_0_142 = rt2_0_142 * (2 + (input[i] > (570+offset2)));
t2_0_142 += (1 - rt2_0_142) * (input[i] > (568+offset2));

int rt2_0_143 = (input[i] > (573+offset2));
int t2_0_143 = rt2_0_143 * (2 + (input[i] > (574+offset2)));
t2_0_143 += (1 - rt2_0_143) * (input[i] > (572+offset2));

int rt2_0_144 = (input[i] > (577+offset2));
int t2_0_144 = rt2_0_144 * (2 + (input[i] > (578+offset2)));
t2_0_144 += (1 - rt2_0_144) * (input[i] > (576+offset2));

int rt2_0_145 = (input[i] > (581+offset2));
int t2_0_145 = rt2_0_145 * (2 + (input[i] > (582+offset2)));
t2_0_145 += (1 - rt2_0_145) * (input[i] > (580+offset2));

int rt2_0_146 = (input[i] > (585+offset2));
int t2_0_146 = rt2_0_146 * (2 + (input[i] > (586+offset2)));
t2_0_146 += (1 - rt2_0_146) * (input[i] > (584+offset2));

int rt2_0_147 = (input[i] > (589+offset2));
int t2_0_147 = rt2_0_147 * (2 + (input[i] > (590+offset2)));
t2_0_147 += (1 - rt2_0_147) * (input[i] > (588+offset2));

int rt2_0_148 = (input[i] > (593+offset2));
int t2_0_148 = rt2_0_148 * (2 + (input[i] > (594+offset2)));
t2_0_148 += (1 - rt2_0_148) * (input[i] > (592+offset2));

int rt2_0_149 = (input[i] > (597+offset2));
int t2_0_149 = rt2_0_149 * (2 + (input[i] > (598+offset2)));
t2_0_149 += (1 - rt2_0_149) * (input[i] > (596+offset2));

int rt2_0_150 = (input[i] > (601+offset2));
int t2_0_150 = rt2_0_150 * (2 + (input[i] > (602+offset2)));
t2_0_150 += (1 - rt2_0_150) * (input[i] > (600+offset2));

int rt2_0_151 = (input[i] > (605+offset2));
int t2_0_151 = rt2_0_151 * (2 + (input[i] > (606+offset2)));
t2_0_151 += (1 - rt2_0_151) * (input[i] > (604+offset2));

int rt2_0_152 = (input[i] > (609+offset2));
int t2_0_152 = rt2_0_152 * (2 + (input[i] > (610+offset2)));
t2_0_152 += (1 - rt2_0_152) * (input[i] > (608+offset2));

int rt2_0_153 = (input[i] > (613+offset2));
int t2_0_153 = rt2_0_153 * (2 + (input[i] > (614+offset2)));
t2_0_153 += (1 - rt2_0_153) * (input[i] > (612+offset2));

int rt2_0_154 = (input[i] > (617+offset2));
int t2_0_154 = rt2_0_154 * (2 + (input[i] > (618+offset2)));
t2_0_154 += (1 - rt2_0_154) * (input[i] > (616+offset2));

int rt2_0_155 = (input[i] > (621+offset2));
int t2_0_155 = rt2_0_155 * (2 + (input[i] > (622+offset2)));
t2_0_155 += (1 - rt2_0_155) * (input[i] > (620+offset2));

int rt2_0_156 = (input[i] > (625+offset2));
int t2_0_156 = rt2_0_156 * (2 + (input[i] > (626+offset2)));
t2_0_156 += (1 - rt2_0_156) * (input[i] > (624+offset2));

int rt2_0_157 = (input[i] > (629+offset2));
int t2_0_157 = rt2_0_157 * (2 + (input[i] > (630+offset2)));
t2_0_157 += (1 - rt2_0_157) * (input[i] > (628+offset2));

int rt2_0_158 = (input[i] > (633+offset2));
int t2_0_158 = rt2_0_158 * (2 + (input[i] > (634+offset2)));
t2_0_158 += (1 - rt2_0_158) * (input[i] > (632+offset2));

int rt2_0_159 = (input[i] > (637+offset2));
int t2_0_159 = rt2_0_159 * (2 + (input[i] > (638+offset2)));
t2_0_159 += (1 - rt2_0_159) * (input[i] > (636+offset2));

int rt2_0_160 = (input[i] > (641+offset2));
int t2_0_160 = rt2_0_160 * (2 + (input[i] > (642+offset2)));
t2_0_160 += (1 - rt2_0_160) * (input[i] > (640+offset2));

int rt2_0_161 = (input[i] > (645+offset2));
int t2_0_161 = rt2_0_161 * (2 + (input[i] > (646+offset2)));
t2_0_161 += (1 - rt2_0_161) * (input[i] > (644+offset2));

int rt2_0_162 = (input[i] > (649+offset2));
int t2_0_162 = rt2_0_162 * (2 + (input[i] > (650+offset2)));
t2_0_162 += (1 - rt2_0_162) * (input[i] > (648+offset2));

int rt2_0_163 = (input[i] > (653+offset2));
int t2_0_163 = rt2_0_163 * (2 + (input[i] > (654+offset2)));
t2_0_163 += (1 - rt2_0_163) * (input[i] > (652+offset2));

int rt2_0_164 = (input[i] > (657+offset2));
int t2_0_164 = rt2_0_164 * (2 + (input[i] > (658+offset2)));
t2_0_164 += (1 - rt2_0_164) * (input[i] > (656+offset2));

int rt2_0_165 = (input[i] > (661+offset2));
int t2_0_165 = rt2_0_165 * (2 + (input[i] > (662+offset2)));
t2_0_165 += (1 - rt2_0_165) * (input[i] > (660+offset2));

int rt2_0_166 = (input[i] > (665+offset2));
int t2_0_166 = rt2_0_166 * (2 + (input[i] > (666+offset2)));
t2_0_166 += (1 - rt2_0_166) * (input[i] > (664+offset2));

int rt2_0_167 = (input[i] > (669+offset2));
int t2_0_167 = rt2_0_167 * (2 + (input[i] > (670+offset2)));
t2_0_167 += (1 - rt2_0_167) * (input[i] > (668+offset2));

int rt2_0_168 = (input[i] > (673+offset2));
int t2_0_168 = rt2_0_168 * (2 + (input[i] > (674+offset2)));
t2_0_168 += (1 - rt2_0_168) * (input[i] > (672+offset2));

int rt2_0_169 = (input[i] > (677+offset2));
int t2_0_169 = rt2_0_169 * (2 + (input[i] > (678+offset2)));
t2_0_169 += (1 - rt2_0_169) * (input[i] > (676+offset2));

int rt2_0_170 = (input[i] > (681+offset2));
int t2_0_170 = rt2_0_170 * (2 + (input[i] > (682+offset2)));
t2_0_170 += (1 - rt2_0_170) * (input[i] > (680+offset2));

int rt2_0_171 = (input[i] > (685+offset2));
int t2_0_171 = rt2_0_171 * (2 + (input[i] > (686+offset2)));
t2_0_171 += (1 - rt2_0_171) * (input[i] > (684+offset2));

int rt2_0_172 = (input[i] > (689+offset2));
int t2_0_172 = rt2_0_172 * (2 + (input[i] > (690+offset2)));
t2_0_172 += (1 - rt2_0_172) * (input[i] > (688+offset2));

int rt2_0_173 = (input[i] > (693+offset2));
int t2_0_173 = rt2_0_173 * (2 + (input[i] > (694+offset2)));
t2_0_173 += (1 - rt2_0_173) * (input[i] > (692+offset2));

int rt2_0_174 = (input[i] > (697+offset2));
int t2_0_174 = rt2_0_174 * (2 + (input[i] > (698+offset2)));
t2_0_174 += (1 - rt2_0_174) * (input[i] > (696+offset2));

int rt2_0_175 = (input[i] > (701+offset2));
int t2_0_175 = rt2_0_175 * (2 + (input[i] > (702+offset2)));
t2_0_175 += (1 - rt2_0_175) * (input[i] > (700+offset2));

int rt2_0_176 = (input[i] > (705+offset2));
int t2_0_176 = rt2_0_176 * (2 + (input[i] > (706+offset2)));
t2_0_176 += (1 - rt2_0_176) * (input[i] > (704+offset2));

int rt2_0_177 = (input[i] > (709+offset2));
int t2_0_177 = rt2_0_177 * (2 + (input[i] > (710+offset2)));
t2_0_177 += (1 - rt2_0_177) * (input[i] > (708+offset2));

int rt2_0_178 = (input[i] > (713+offset2));
int t2_0_178 = rt2_0_178 * (2 + (input[i] > (714+offset2)));
t2_0_178 += (1 - rt2_0_178) * (input[i] > (712+offset2));

int rt2_0_179 = (input[i] > (717+offset2));
int t2_0_179 = rt2_0_179 * (2 + (input[i] > (718+offset2)));
t2_0_179 += (1 - rt2_0_179) * (input[i] > (716+offset2));

int rt2_0_180 = (input[i] > (721+offset2));
int t2_0_180 = rt2_0_180 * (2 + (input[i] > (722+offset2)));
t2_0_180 += (1 - rt2_0_180) * (input[i] > (720+offset2));

int rt2_0_181 = (input[i] > (725+offset2));
int t2_0_181 = rt2_0_181 * (2 + (input[i] > (726+offset2)));
t2_0_181 += (1 - rt2_0_181) * (input[i] > (724+offset2));

int rt2_0_182 = (input[i] > (729+offset2));
int t2_0_182 = rt2_0_182 * (2 + (input[i] > (730+offset2)));
t2_0_182 += (1 - rt2_0_182) * (input[i] > (728+offset2));

int rt2_0_183 = (input[i] > (733+offset2));
int t2_0_183 = rt2_0_183 * (2 + (input[i] > (734+offset2)));
t2_0_183 += (1 - rt2_0_183) * (input[i] > (732+offset2));

int rt2_0_184 = (input[i] > (737+offset2));
int t2_0_184 = rt2_0_184 * (2 + (input[i] > (738+offset2)));
t2_0_184 += (1 - rt2_0_184) * (input[i] > (736+offset2));

int rt2_0_185 = (input[i] > (741+offset2));
int t2_0_185 = rt2_0_185 * (2 + (input[i] > (742+offset2)));
t2_0_185 += (1 - rt2_0_185) * (input[i] > (740+offset2));

int rt2_0_186 = (input[i] > (745+offset2));
int t2_0_186 = rt2_0_186 * (2 + (input[i] > (746+offset2)));
t2_0_186 += (1 - rt2_0_186) * (input[i] > (744+offset2));

int rt2_0_187 = (input[i] > (749+offset2));
int t2_0_187 = rt2_0_187 * (2 + (input[i] > (750+offset2)));
t2_0_187 += (1 - rt2_0_187) * (input[i] > (748+offset2));

int rt2_0_188 = (input[i] > (753+offset2));
int t2_0_188 = rt2_0_188 * (2 + (input[i] > (754+offset2)));
t2_0_188 += (1 - rt2_0_188) * (input[i] > (752+offset2));

int rt2_0_189 = (input[i] > (757+offset2));
int t2_0_189 = rt2_0_189 * (2 + (input[i] > (758+offset2)));
t2_0_189 += (1 - rt2_0_189) * (input[i] > (756+offset2));

int rt2_0_190 = (input[i] > (761+offset2));
int t2_0_190 = rt2_0_190 * (2 + (input[i] > (762+offset2)));
t2_0_190 += (1 - rt2_0_190) * (input[i] > (760+offset2));

int rt2_0_191 = (input[i] > (765+offset2));
int t2_0_191 = rt2_0_191 * (2 + (input[i] > (766+offset2)));
t2_0_191 += (1 - rt2_0_191) * (input[i] > (764+offset2));

int rt2_0_192 = (input[i] > (769+offset2));
int t2_0_192 = rt2_0_192 * (2 + (input[i] > (770+offset2)));
t2_0_192 += (1 - rt2_0_192) * (input[i] > (768+offset2));

int rt2_0_193 = (input[i] > (773+offset2));
int t2_0_193 = rt2_0_193 * (2 + (input[i] > (774+offset2)));
t2_0_193 += (1 - rt2_0_193) * (input[i] > (772+offset2));

int rt2_0_194 = (input[i] > (777+offset2));
int t2_0_194 = rt2_0_194 * (2 + (input[i] > (778+offset2)));
t2_0_194 += (1 - rt2_0_194) * (input[i] > (776+offset2));

int rt2_0_195 = (input[i] > (781+offset2));
int t2_0_195 = rt2_0_195 * (2 + (input[i] > (782+offset2)));
t2_0_195 += (1 - rt2_0_195) * (input[i] > (780+offset2));

int rt2_0_196 = (input[i] > (785+offset2));
int t2_0_196 = rt2_0_196 * (2 + (input[i] > (786+offset2)));
t2_0_196 += (1 - rt2_0_196) * (input[i] > (784+offset2));

int rt2_0_197 = (input[i] > (789+offset2));
int t2_0_197 = rt2_0_197 * (2 + (input[i] > (790+offset2)));
t2_0_197 += (1 - rt2_0_197) * (input[i] > (788+offset2));

int rt2_0_198 = (input[i] > (793+offset2));
int t2_0_198 = rt2_0_198 * (2 + (input[i] > (794+offset2)));
t2_0_198 += (1 - rt2_0_198) * (input[i] > (792+offset2));

int rt2_0_199 = (input[i] > (797+offset2));
int t2_0_199 = rt2_0_199 * (2 + (input[i] > (798+offset2)));
t2_0_199 += (1 - rt2_0_199) * (input[i] > (796+offset2));

int rt2_0_200 = (input[i] > (801+offset2));
int t2_0_200 = rt2_0_200 * (2 + (input[i] > (802+offset2)));
t2_0_200 += (1 - rt2_0_200) * (input[i] > (800+offset2));

int rt2_0_201 = (input[i] > (805+offset2));
int t2_0_201 = rt2_0_201 * (2 + (input[i] > (806+offset2)));
t2_0_201 += (1 - rt2_0_201) * (input[i] > (804+offset2));

int rt2_0_202 = (input[i] > (809+offset2));
int t2_0_202 = rt2_0_202 * (2 + (input[i] > (810+offset2)));
t2_0_202 += (1 - rt2_0_202) * (input[i] > (808+offset2));

int rt2_0_203 = (input[i] > (813+offset2));
int t2_0_203 = rt2_0_203 * (2 + (input[i] > (814+offset2)));
t2_0_203 += (1 - rt2_0_203) * (input[i] > (812+offset2));

int rt2_0_204 = (input[i] > (817+offset2));
int t2_0_204 = rt2_0_204 * (2 + (input[i] > (818+offset2)));
t2_0_204 += (1 - rt2_0_204) * (input[i] > (816+offset2));

int rt2_0_205 = (input[i] > (821+offset2));
int t2_0_205 = rt2_0_205 * (2 + (input[i] > (822+offset2)));
t2_0_205 += (1 - rt2_0_205) * (input[i] > (820+offset2));

int rt2_0_206 = (input[i] > (825+offset2));
int t2_0_206 = rt2_0_206 * (2 + (input[i] > (826+offset2)));
t2_0_206 += (1 - rt2_0_206) * (input[i] > (824+offset2));

int rt2_0_207 = (input[i] > (829+offset2));
int t2_0_207 = rt2_0_207 * (2 + (input[i] > (830+offset2)));
t2_0_207 += (1 - rt2_0_207) * (input[i] > (828+offset2));

int rt2_0_208 = (input[i] > (833+offset2));
int t2_0_208 = rt2_0_208 * (2 + (input[i] > (834+offset2)));
t2_0_208 += (1 - rt2_0_208) * (input[i] > (832+offset2));

int rt2_0_209 = (input[i] > (837+offset2));
int t2_0_209 = rt2_0_209 * (2 + (input[i] > (838+offset2)));
t2_0_209 += (1 - rt2_0_209) * (input[i] > (836+offset2));

int rt2_0_210 = (input[i] > (841+offset2));
int t2_0_210 = rt2_0_210 * (2 + (input[i] > (842+offset2)));
t2_0_210 += (1 - rt2_0_210) * (input[i] > (840+offset2));

int rt2_0_211 = (input[i] > (845+offset2));
int t2_0_211 = rt2_0_211 * (2 + (input[i] > (846+offset2)));
t2_0_211 += (1 - rt2_0_211) * (input[i] > (844+offset2));

int rt2_0_212 = (input[i] > (849+offset2));
int t2_0_212 = rt2_0_212 * (2 + (input[i] > (850+offset2)));
t2_0_212 += (1 - rt2_0_212) * (input[i] > (848+offset2));

int rt2_0_213 = (input[i] > (853+offset2));
int t2_0_213 = rt2_0_213 * (2 + (input[i] > (854+offset2)));
t2_0_213 += (1 - rt2_0_213) * (input[i] > (852+offset2));

int rt2_0_214 = (input[i] > (857+offset2));
int t2_0_214 = rt2_0_214 * (2 + (input[i] > (858+offset2)));
t2_0_214 += (1 - rt2_0_214) * (input[i] > (856+offset2));

int rt2_0_215 = (input[i] > (861+offset2));
int t2_0_215 = rt2_0_215 * (2 + (input[i] > (862+offset2)));
t2_0_215 += (1 - rt2_0_215) * (input[i] > (860+offset2));

int rt2_0_216 = (input[i] > (865+offset2));
int t2_0_216 = rt2_0_216 * (2 + (input[i] > (866+offset2)));
t2_0_216 += (1 - rt2_0_216) * (input[i] > (864+offset2));

int rt2_0_217 = (input[i] > (869+offset2));
int t2_0_217 = rt2_0_217 * (2 + (input[i] > (870+offset2)));
t2_0_217 += (1 - rt2_0_217) * (input[i] > (868+offset2));

int rt2_0_218 = (input[i] > (873+offset2));
int t2_0_218 = rt2_0_218 * (2 + (input[i] > (874+offset2)));
t2_0_218 += (1 - rt2_0_218) * (input[i] > (872+offset2));

int rt2_0_219 = (input[i] > (877+offset2));
int t2_0_219 = rt2_0_219 * (2 + (input[i] > (878+offset2)));
t2_0_219 += (1 - rt2_0_219) * (input[i] > (876+offset2));

int rt2_0_220 = (input[i] > (881+offset2));
int t2_0_220 = rt2_0_220 * (2 + (input[i] > (882+offset2)));
t2_0_220 += (1 - rt2_0_220) * (input[i] > (880+offset2));

int rt2_0_221 = (input[i] > (885+offset2));
int t2_0_221 = rt2_0_221 * (2 + (input[i] > (886+offset2)));
t2_0_221 += (1 - rt2_0_221) * (input[i] > (884+offset2));

int rt2_0_222 = (input[i] > (889+offset2));
int t2_0_222 = rt2_0_222 * (2 + (input[i] > (890+offset2)));
t2_0_222 += (1 - rt2_0_222) * (input[i] > (888+offset2));

int rt2_0_223 = (input[i] > (893+offset2));
int t2_0_223 = rt2_0_223 * (2 + (input[i] > (894+offset2)));
t2_0_223 += (1 - rt2_0_223) * (input[i] > (892+offset2));

int rt2_0_224 = (input[i] > (897+offset2));
int t2_0_224 = rt2_0_224 * (2 + (input[i] > (898+offset2)));
t2_0_224 += (1 - rt2_0_224) * (input[i] > (896+offset2));

int rt2_0_225 = (input[i] > (901+offset2));
int t2_0_225 = rt2_0_225 * (2 + (input[i] > (902+offset2)));
t2_0_225 += (1 - rt2_0_225) * (input[i] > (900+offset2));

int rt2_0_226 = (input[i] > (905+offset2));
int t2_0_226 = rt2_0_226 * (2 + (input[i] > (906+offset2)));
t2_0_226 += (1 - rt2_0_226) * (input[i] > (904+offset2));

int rt2_0_227 = (input[i] > (909+offset2));
int t2_0_227 = rt2_0_227 * (2 + (input[i] > (910+offset2)));
t2_0_227 += (1 - rt2_0_227) * (input[i] > (908+offset2));

int rt2_0_228 = (input[i] > (913+offset2));
int t2_0_228 = rt2_0_228 * (2 + (input[i] > (914+offset2)));
t2_0_228 += (1 - rt2_0_228) * (input[i] > (912+offset2));

int rt2_0_229 = (input[i] > (917+offset2));
int t2_0_229 = rt2_0_229 * (2 + (input[i] > (918+offset2)));
t2_0_229 += (1 - rt2_0_229) * (input[i] > (916+offset2));

int rt2_0_230 = (input[i] > (921+offset2));
int t2_0_230 = rt2_0_230 * (2 + (input[i] > (922+offset2)));
t2_0_230 += (1 - rt2_0_230) * (input[i] > (920+offset2));

int rt2_0_231 = (input[i] > (925+offset2));
int t2_0_231 = rt2_0_231 * (2 + (input[i] > (926+offset2)));
t2_0_231 += (1 - rt2_0_231) * (input[i] > (924+offset2));

int rt2_0_232 = (input[i] > (929+offset2));
int t2_0_232 = rt2_0_232 * (2 + (input[i] > (930+offset2)));
t2_0_232 += (1 - rt2_0_232) * (input[i] > (928+offset2));

int rt2_0_233 = (input[i] > (933+offset2));
int t2_0_233 = rt2_0_233 * (2 + (input[i] > (934+offset2)));
t2_0_233 += (1 - rt2_0_233) * (input[i] > (932+offset2));

int rt2_0_234 = (input[i] > (937+offset2));
int t2_0_234 = rt2_0_234 * (2 + (input[i] > (938+offset2)));
t2_0_234 += (1 - rt2_0_234) * (input[i] > (936+offset2));

int rt2_0_235 = (input[i] > (941+offset2));
int t2_0_235 = rt2_0_235 * (2 + (input[i] > (942+offset2)));
t2_0_235 += (1 - rt2_0_235) * (input[i] > (940+offset2));

int rt2_0_236 = (input[i] > (945+offset2));
int t2_0_236 = rt2_0_236 * (2 + (input[i] > (946+offset2)));
t2_0_236 += (1 - rt2_0_236) * (input[i] > (944+offset2));

int rt2_0_237 = (input[i] > (949+offset2));
int t2_0_237 = rt2_0_237 * (2 + (input[i] > (950+offset2)));
t2_0_237 += (1 - rt2_0_237) * (input[i] > (948+offset2));

int rt2_0_238 = (input[i] > (953+offset2));
int t2_0_238 = rt2_0_238 * (2 + (input[i] > (954+offset2)));
t2_0_238 += (1 - rt2_0_238) * (input[i] > (952+offset2));

int rt2_0_239 = (input[i] > (957+offset2));
int t2_0_239 = rt2_0_239 * (2 + (input[i] > (958+offset2)));
t2_0_239 += (1 - rt2_0_239) * (input[i] > (956+offset2));

int rt2_0_240 = (input[i] > (961+offset2));
int t2_0_240 = rt2_0_240 * (2 + (input[i] > (962+offset2)));
t2_0_240 += (1 - rt2_0_240) * (input[i] > (960+offset2));

int rt2_0_241 = (input[i] > (965+offset2));
int t2_0_241 = rt2_0_241 * (2 + (input[i] > (966+offset2)));
t2_0_241 += (1 - rt2_0_241) * (input[i] > (964+offset2));

int rt2_0_242 = (input[i] > (969+offset2));
int t2_0_242 = rt2_0_242 * (2 + (input[i] > (970+offset2)));
t2_0_242 += (1 - rt2_0_242) * (input[i] > (968+offset2));

int rt2_0_243 = (input[i] > (973+offset2));
int t2_0_243 = rt2_0_243 * (2 + (input[i] > (974+offset2)));
t2_0_243 += (1 - rt2_0_243) * (input[i] > (972+offset2));

int rt2_0_244 = (input[i] > (977+offset2));
int t2_0_244 = rt2_0_244 * (2 + (input[i] > (978+offset2)));
t2_0_244 += (1 - rt2_0_244) * (input[i] > (976+offset2));

int rt2_0_245 = (input[i] > (981+offset2));
int t2_0_245 = rt2_0_245 * (2 + (input[i] > (982+offset2)));
t2_0_245 += (1 - rt2_0_245) * (input[i] > (980+offset2));

int rt2_0_246 = (input[i] > (985+offset2));
int t2_0_246 = rt2_0_246 * (2 + (input[i] > (986+offset2)));
t2_0_246 += (1 - rt2_0_246) * (input[i] > (984+offset2));

int rt2_0_247 = (input[i] > (989+offset2));
int t2_0_247 = rt2_0_247 * (2 + (input[i] > (990+offset2)));
t2_0_247 += (1 - rt2_0_247) * (input[i] > (988+offset2));

int rt2_0_248 = (input[i] > (993+offset2));
int t2_0_248 = rt2_0_248 * (2 + (input[i] > (994+offset2)));
t2_0_248 += (1 - rt2_0_248) * (input[i] > (992+offset2));

int rt2_0_249 = (input[i] > (997+offset2));
int t2_0_249 = rt2_0_249 * (2 + (input[i] > (998+offset2)));
t2_0_249 += (1 - rt2_0_249) * (input[i] > (996+offset2));

int rt2_0_250 = (input[i] > (1001+offset2));
int t2_0_250 = rt2_0_250 * (2 + (input[i] > (1002+offset2)));
t2_0_250 += (1 - rt2_0_250) * (input[i] > (1000+offset2));

int rt2_0_251 = (input[i] > (1005+offset2));
int t2_0_251 = rt2_0_251 * (2 + (input[i] > (1006+offset2)));
t2_0_251 += (1 - rt2_0_251) * (input[i] > (1004+offset2));

int rt2_0_252 = (input[i] > (1009+offset2));
int t2_0_252 = rt2_0_252 * (2 + (input[i] > (1010+offset2)));
t2_0_252 += (1 - rt2_0_252) * (input[i] > (1008+offset2));

int rt2_0_253 = (input[i] > (1013+offset2));
int t2_0_253 = rt2_0_253 * (2 + (input[i] > (1014+offset2)));
t2_0_253 += (1 - rt2_0_253) * (input[i] > (1012+offset2));

int rt2_0_254 = (input[i] > (1017+offset2));
int t2_0_254 = rt2_0_254 * (2 + (input[i] > (1018+offset2)));
t2_0_254 += (1 - rt2_0_254) * (input[i] > (1016+offset2));

int rt2_0_255 = (input[i] > (1021+offset2));
int t2_0_255 = rt2_0_255 * (2 + (input[i] > (1022+offset2)));
t2_0_255 += (1 - rt2_0_255) * (input[i] > (1020+offset2));

int rt2_1_0 = (input[i] > (3+offset2));
int t2_1_0 = rt2_1_0 * (4 + t2_0_1);
t2_1_0 += (1 - rt2_1_0) * t2_0_0;

int rt2_1_1 = (input[i] > (11+offset2));
int t2_1_1 = rt2_1_1 * (4 + t2_0_3);
t2_1_1 += (1 - rt2_1_1) * t2_0_2;

int rt2_1_2 = (input[i] > (19+offset2));
int t2_1_2 = rt2_1_2 * (4 + t2_0_5);
t2_1_2 += (1 - rt2_1_2) * t2_0_4;

int rt2_1_3 = (input[i] > (27+offset2));
int t2_1_3 = rt2_1_3 * (4 + t2_0_7);
t2_1_3 += (1 - rt2_1_3) * t2_0_6;

int rt2_1_4 = (input[i] > (35+offset2));
int t2_1_4 = rt2_1_4 * (4 + t2_0_9);
t2_1_4 += (1 - rt2_1_4) * t2_0_8;

int rt2_1_5 = (input[i] > (43+offset2));
int t2_1_5 = rt2_1_5 * (4 + t2_0_11);
t2_1_5 += (1 - rt2_1_5) * t2_0_10;

int rt2_1_6 = (input[i] > (51+offset2));
int t2_1_6 = rt2_1_6 * (4 + t2_0_13);
t2_1_6 += (1 - rt2_1_6) * t2_0_12;

int rt2_1_7 = (input[i] > (59+offset2));
int t2_1_7 = rt2_1_7 * (4 + t2_0_15);
t2_1_7 += (1 - rt2_1_7) * t2_0_14;

int rt2_1_8 = (input[i] > (67+offset2));
int t2_1_8 = rt2_1_8 * (4 + t2_0_17);
t2_1_8 += (1 - rt2_1_8) * t2_0_16;

int rt2_1_9 = (input[i] > (75+offset2));
int t2_1_9 = rt2_1_9 * (4 + t2_0_19);
t2_1_9 += (1 - rt2_1_9) * t2_0_18;

int rt2_1_10 = (input[i] > (83+offset2));
int t2_1_10 = rt2_1_10 * (4 + t2_0_21);
t2_1_10 += (1 - rt2_1_10) * t2_0_20;

int rt2_1_11 = (input[i] > (91+offset2));
int t2_1_11 = rt2_1_11 * (4 + t2_0_23);
t2_1_11 += (1 - rt2_1_11) * t2_0_22;

int rt2_1_12 = (input[i] > (99+offset2));
int t2_1_12 = rt2_1_12 * (4 + t2_0_25);
t2_1_12 += (1 - rt2_1_12) * t2_0_24;

int rt2_1_13 = (input[i] > (107+offset2));
int t2_1_13 = rt2_1_13 * (4 + t2_0_27);
t2_1_13 += (1 - rt2_1_13) * t2_0_26;

int rt2_1_14 = (input[i] > (115+offset2));
int t2_1_14 = rt2_1_14 * (4 + t2_0_29);
t2_1_14 += (1 - rt2_1_14) * t2_0_28;

int rt2_1_15 = (input[i] > (123+offset2));
int t2_1_15 = rt2_1_15 * (4 + t2_0_31);
t2_1_15 += (1 - rt2_1_15) * t2_0_30;

int rt2_1_16 = (input[i] > (131+offset2));
int t2_1_16 = rt2_1_16 * (4 + t2_0_33);
t2_1_16 += (1 - rt2_1_16) * t2_0_32;

int rt2_1_17 = (input[i] > (139+offset2));
int t2_1_17 = rt2_1_17 * (4 + t2_0_35);
t2_1_17 += (1 - rt2_1_17) * t2_0_34;

int rt2_1_18 = (input[i] > (147+offset2));
int t2_1_18 = rt2_1_18 * (4 + t2_0_37);
t2_1_18 += (1 - rt2_1_18) * t2_0_36;

int rt2_1_19 = (input[i] > (155+offset2));
int t2_1_19 = rt2_1_19 * (4 + t2_0_39);
t2_1_19 += (1 - rt2_1_19) * t2_0_38;

int rt2_1_20 = (input[i] > (163+offset2));
int t2_1_20 = rt2_1_20 * (4 + t2_0_41);
t2_1_20 += (1 - rt2_1_20) * t2_0_40;

int rt2_1_21 = (input[i] > (171+offset2));
int t2_1_21 = rt2_1_21 * (4 + t2_0_43);
t2_1_21 += (1 - rt2_1_21) * t2_0_42;

int rt2_1_22 = (input[i] > (179+offset2));
int t2_1_22 = rt2_1_22 * (4 + t2_0_45);
t2_1_22 += (1 - rt2_1_22) * t2_0_44;

int rt2_1_23 = (input[i] > (187+offset2));
int t2_1_23 = rt2_1_23 * (4 + t2_0_47);
t2_1_23 += (1 - rt2_1_23) * t2_0_46;

int rt2_1_24 = (input[i] > (195+offset2));
int t2_1_24 = rt2_1_24 * (4 + t2_0_49);
t2_1_24 += (1 - rt2_1_24) * t2_0_48;

int rt2_1_25 = (input[i] > (203+offset2));
int t2_1_25 = rt2_1_25 * (4 + t2_0_51);
t2_1_25 += (1 - rt2_1_25) * t2_0_50;

int rt2_1_26 = (input[i] > (211+offset2));
int t2_1_26 = rt2_1_26 * (4 + t2_0_53);
t2_1_26 += (1 - rt2_1_26) * t2_0_52;

int rt2_1_27 = (input[i] > (219+offset2));
int t2_1_27 = rt2_1_27 * (4 + t2_0_55);
t2_1_27 += (1 - rt2_1_27) * t2_0_54;

int rt2_1_28 = (input[i] > (227+offset2));
int t2_1_28 = rt2_1_28 * (4 + t2_0_57);
t2_1_28 += (1 - rt2_1_28) * t2_0_56;

int rt2_1_29 = (input[i] > (235+offset2));
int t2_1_29 = rt2_1_29 * (4 + t2_0_59);
t2_1_29 += (1 - rt2_1_29) * t2_0_58;

int rt2_1_30 = (input[i] > (243+offset2));
int t2_1_30 = rt2_1_30 * (4 + t2_0_61);
t2_1_30 += (1 - rt2_1_30) * t2_0_60;

int rt2_1_31 = (input[i] > (251+offset2));
int t2_1_31 = rt2_1_31 * (4 + t2_0_63);
t2_1_31 += (1 - rt2_1_31) * t2_0_62;

int rt2_1_32 = (input[i] > (259+offset2));
int t2_1_32 = rt2_1_32 * (4 + t2_0_65);
t2_1_32 += (1 - rt2_1_32) * t2_0_64;

int rt2_1_33 = (input[i] > (267+offset2));
int t2_1_33 = rt2_1_33 * (4 + t2_0_67);
t2_1_33 += (1 - rt2_1_33) * t2_0_66;

int rt2_1_34 = (input[i] > (275+offset2));
int t2_1_34 = rt2_1_34 * (4 + t2_0_69);
t2_1_34 += (1 - rt2_1_34) * t2_0_68;

int rt2_1_35 = (input[i] > (283+offset2));
int t2_1_35 = rt2_1_35 * (4 + t2_0_71);
t2_1_35 += (1 - rt2_1_35) * t2_0_70;

int rt2_1_36 = (input[i] > (291+offset2));
int t2_1_36 = rt2_1_36 * (4 + t2_0_73);
t2_1_36 += (1 - rt2_1_36) * t2_0_72;

int rt2_1_37 = (input[i] > (299+offset2));
int t2_1_37 = rt2_1_37 * (4 + t2_0_75);
t2_1_37 += (1 - rt2_1_37) * t2_0_74;

int rt2_1_38 = (input[i] > (307+offset2));
int t2_1_38 = rt2_1_38 * (4 + t2_0_77);
t2_1_38 += (1 - rt2_1_38) * t2_0_76;

int rt2_1_39 = (input[i] > (315+offset2));
int t2_1_39 = rt2_1_39 * (4 + t2_0_79);
t2_1_39 += (1 - rt2_1_39) * t2_0_78;

int rt2_1_40 = (input[i] > (323+offset2));
int t2_1_40 = rt2_1_40 * (4 + t2_0_81);
t2_1_40 += (1 - rt2_1_40) * t2_0_80;

int rt2_1_41 = (input[i] > (331+offset2));
int t2_1_41 = rt2_1_41 * (4 + t2_0_83);
t2_1_41 += (1 - rt2_1_41) * t2_0_82;

int rt2_1_42 = (input[i] > (339+offset2));
int t2_1_42 = rt2_1_42 * (4 + t2_0_85);
t2_1_42 += (1 - rt2_1_42) * t2_0_84;

int rt2_1_43 = (input[i] > (347+offset2));
int t2_1_43 = rt2_1_43 * (4 + t2_0_87);
t2_1_43 += (1 - rt2_1_43) * t2_0_86;

int rt2_1_44 = (input[i] > (355+offset2));
int t2_1_44 = rt2_1_44 * (4 + t2_0_89);
t2_1_44 += (1 - rt2_1_44) * t2_0_88;

int rt2_1_45 = (input[i] > (363+offset2));
int t2_1_45 = rt2_1_45 * (4 + t2_0_91);
t2_1_45 += (1 - rt2_1_45) * t2_0_90;

int rt2_1_46 = (input[i] > (371+offset2));
int t2_1_46 = rt2_1_46 * (4 + t2_0_93);
t2_1_46 += (1 - rt2_1_46) * t2_0_92;

int rt2_1_47 = (input[i] > (379+offset2));
int t2_1_47 = rt2_1_47 * (4 + t2_0_95);
t2_1_47 += (1 - rt2_1_47) * t2_0_94;

int rt2_1_48 = (input[i] > (387+offset2));
int t2_1_48 = rt2_1_48 * (4 + t2_0_97);
t2_1_48 += (1 - rt2_1_48) * t2_0_96;

int rt2_1_49 = (input[i] > (395+offset2));
int t2_1_49 = rt2_1_49 * (4 + t2_0_99);
t2_1_49 += (1 - rt2_1_49) * t2_0_98;

int rt2_1_50 = (input[i] > (403+offset2));
int t2_1_50 = rt2_1_50 * (4 + t2_0_101);
t2_1_50 += (1 - rt2_1_50) * t2_0_100;

int rt2_1_51 = (input[i] > (411+offset2));
int t2_1_51 = rt2_1_51 * (4 + t2_0_103);
t2_1_51 += (1 - rt2_1_51) * t2_0_102;

int rt2_1_52 = (input[i] > (419+offset2));
int t2_1_52 = rt2_1_52 * (4 + t2_0_105);
t2_1_52 += (1 - rt2_1_52) * t2_0_104;

int rt2_1_53 = (input[i] > (427+offset2));
int t2_1_53 = rt2_1_53 * (4 + t2_0_107);
t2_1_53 += (1 - rt2_1_53) * t2_0_106;

int rt2_1_54 = (input[i] > (435+offset2));
int t2_1_54 = rt2_1_54 * (4 + t2_0_109);
t2_1_54 += (1 - rt2_1_54) * t2_0_108;

int rt2_1_55 = (input[i] > (443+offset2));
int t2_1_55 = rt2_1_55 * (4 + t2_0_111);
t2_1_55 += (1 - rt2_1_55) * t2_0_110;

int rt2_1_56 = (input[i] > (451+offset2));
int t2_1_56 = rt2_1_56 * (4 + t2_0_113);
t2_1_56 += (1 - rt2_1_56) * t2_0_112;

int rt2_1_57 = (input[i] > (459+offset2));
int t2_1_57 = rt2_1_57 * (4 + t2_0_115);
t2_1_57 += (1 - rt2_1_57) * t2_0_114;

int rt2_1_58 = (input[i] > (467+offset2));
int t2_1_58 = rt2_1_58 * (4 + t2_0_117);
t2_1_58 += (1 - rt2_1_58) * t2_0_116;

int rt2_1_59 = (input[i] > (475+offset2));
int t2_1_59 = rt2_1_59 * (4 + t2_0_119);
t2_1_59 += (1 - rt2_1_59) * t2_0_118;

int rt2_1_60 = (input[i] > (483+offset2));
int t2_1_60 = rt2_1_60 * (4 + t2_0_121);
t2_1_60 += (1 - rt2_1_60) * t2_0_120;

int rt2_1_61 = (input[i] > (491+offset2));
int t2_1_61 = rt2_1_61 * (4 + t2_0_123);
t2_1_61 += (1 - rt2_1_61) * t2_0_122;

int rt2_1_62 = (input[i] > (499+offset2));
int t2_1_62 = rt2_1_62 * (4 + t2_0_125);
t2_1_62 += (1 - rt2_1_62) * t2_0_124;

int rt2_1_63 = (input[i] > (507+offset2));
int t2_1_63 = rt2_1_63 * (4 + t2_0_127);
t2_1_63 += (1 - rt2_1_63) * t2_0_126;

int rt2_1_64 = (input[i] > (515+offset2));
int t2_1_64 = rt2_1_64 * (4 + t2_0_129);
t2_1_64 += (1 - rt2_1_64) * t2_0_128;

int rt2_1_65 = (input[i] > (523+offset2));
int t2_1_65 = rt2_1_65 * (4 + t2_0_131);
t2_1_65 += (1 - rt2_1_65) * t2_0_130;

int rt2_1_66 = (input[i] > (531+offset2));
int t2_1_66 = rt2_1_66 * (4 + t2_0_133);
t2_1_66 += (1 - rt2_1_66) * t2_0_132;

int rt2_1_67 = (input[i] > (539+offset2));
int t2_1_67 = rt2_1_67 * (4 + t2_0_135);
t2_1_67 += (1 - rt2_1_67) * t2_0_134;

int rt2_1_68 = (input[i] > (547+offset2));
int t2_1_68 = rt2_1_68 * (4 + t2_0_137);
t2_1_68 += (1 - rt2_1_68) * t2_0_136;

int rt2_1_69 = (input[i] > (555+offset2));
int t2_1_69 = rt2_1_69 * (4 + t2_0_139);
t2_1_69 += (1 - rt2_1_69) * t2_0_138;

int rt2_1_70 = (input[i] > (563+offset2));
int t2_1_70 = rt2_1_70 * (4 + t2_0_141);
t2_1_70 += (1 - rt2_1_70) * t2_0_140;

int rt2_1_71 = (input[i] > (571+offset2));
int t2_1_71 = rt2_1_71 * (4 + t2_0_143);
t2_1_71 += (1 - rt2_1_71) * t2_0_142;

int rt2_1_72 = (input[i] > (579+offset2));
int t2_1_72 = rt2_1_72 * (4 + t2_0_145);
t2_1_72 += (1 - rt2_1_72) * t2_0_144;

int rt2_1_73 = (input[i] > (587+offset2));
int t2_1_73 = rt2_1_73 * (4 + t2_0_147);
t2_1_73 += (1 - rt2_1_73) * t2_0_146;

int rt2_1_74 = (input[i] > (595+offset2));
int t2_1_74 = rt2_1_74 * (4 + t2_0_149);
t2_1_74 += (1 - rt2_1_74) * t2_0_148;

int rt2_1_75 = (input[i] > (603+offset2));
int t2_1_75 = rt2_1_75 * (4 + t2_0_151);
t2_1_75 += (1 - rt2_1_75) * t2_0_150;

int rt2_1_76 = (input[i] > (611+offset2));
int t2_1_76 = rt2_1_76 * (4 + t2_0_153);
t2_1_76 += (1 - rt2_1_76) * t2_0_152;

int rt2_1_77 = (input[i] > (619+offset2));
int t2_1_77 = rt2_1_77 * (4 + t2_0_155);
t2_1_77 += (1 - rt2_1_77) * t2_0_154;

int rt2_1_78 = (input[i] > (627+offset2));
int t2_1_78 = rt2_1_78 * (4 + t2_0_157);
t2_1_78 += (1 - rt2_1_78) * t2_0_156;

int rt2_1_79 = (input[i] > (635+offset2));
int t2_1_79 = rt2_1_79 * (4 + t2_0_159);
t2_1_79 += (1 - rt2_1_79) * t2_0_158;

int rt2_1_80 = (input[i] > (643+offset2));
int t2_1_80 = rt2_1_80 * (4 + t2_0_161);
t2_1_80 += (1 - rt2_1_80) * t2_0_160;

int rt2_1_81 = (input[i] > (651+offset2));
int t2_1_81 = rt2_1_81 * (4 + t2_0_163);
t2_1_81 += (1 - rt2_1_81) * t2_0_162;

int rt2_1_82 = (input[i] > (659+offset2));
int t2_1_82 = rt2_1_82 * (4 + t2_0_165);
t2_1_82 += (1 - rt2_1_82) * t2_0_164;

int rt2_1_83 = (input[i] > (667+offset2));
int t2_1_83 = rt2_1_83 * (4 + t2_0_167);
t2_1_83 += (1 - rt2_1_83) * t2_0_166;

int rt2_1_84 = (input[i] > (675+offset2));
int t2_1_84 = rt2_1_84 * (4 + t2_0_169);
t2_1_84 += (1 - rt2_1_84) * t2_0_168;

int rt2_1_85 = (input[i] > (683+offset2));
int t2_1_85 = rt2_1_85 * (4 + t2_0_171);
t2_1_85 += (1 - rt2_1_85) * t2_0_170;

int rt2_1_86 = (input[i] > (691+offset2));
int t2_1_86 = rt2_1_86 * (4 + t2_0_173);
t2_1_86 += (1 - rt2_1_86) * t2_0_172;

int rt2_1_87 = (input[i] > (699+offset2));
int t2_1_87 = rt2_1_87 * (4 + t2_0_175);
t2_1_87 += (1 - rt2_1_87) * t2_0_174;

int rt2_1_88 = (input[i] > (707+offset2));
int t2_1_88 = rt2_1_88 * (4 + t2_0_177);
t2_1_88 += (1 - rt2_1_88) * t2_0_176;

int rt2_1_89 = (input[i] > (715+offset2));
int t2_1_89 = rt2_1_89 * (4 + t2_0_179);
t2_1_89 += (1 - rt2_1_89) * t2_0_178;

int rt2_1_90 = (input[i] > (723+offset2));
int t2_1_90 = rt2_1_90 * (4 + t2_0_181);
t2_1_90 += (1 - rt2_1_90) * t2_0_180;

int rt2_1_91 = (input[i] > (731+offset2));
int t2_1_91 = rt2_1_91 * (4 + t2_0_183);
t2_1_91 += (1 - rt2_1_91) * t2_0_182;

int rt2_1_92 = (input[i] > (739+offset2));
int t2_1_92 = rt2_1_92 * (4 + t2_0_185);
t2_1_92 += (1 - rt2_1_92) * t2_0_184;

int rt2_1_93 = (input[i] > (747+offset2));
int t2_1_93 = rt2_1_93 * (4 + t2_0_187);
t2_1_93 += (1 - rt2_1_93) * t2_0_186;

int rt2_1_94 = (input[i] > (755+offset2));
int t2_1_94 = rt2_1_94 * (4 + t2_0_189);
t2_1_94 += (1 - rt2_1_94) * t2_0_188;

int rt2_1_95 = (input[i] > (763+offset2));
int t2_1_95 = rt2_1_95 * (4 + t2_0_191);
t2_1_95 += (1 - rt2_1_95) * t2_0_190;

int rt2_1_96 = (input[i] > (771+offset2));
int t2_1_96 = rt2_1_96 * (4 + t2_0_193);
t2_1_96 += (1 - rt2_1_96) * t2_0_192;

int rt2_1_97 = (input[i] > (779+offset2));
int t2_1_97 = rt2_1_97 * (4 + t2_0_195);
t2_1_97 += (1 - rt2_1_97) * t2_0_194;

int rt2_1_98 = (input[i] > (787+offset2));
int t2_1_98 = rt2_1_98 * (4 + t2_0_197);
t2_1_98 += (1 - rt2_1_98) * t2_0_196;

int rt2_1_99 = (input[i] > (795+offset2));
int t2_1_99 = rt2_1_99 * (4 + t2_0_199);
t2_1_99 += (1 - rt2_1_99) * t2_0_198;

int rt2_1_100 = (input[i] > (803+offset2));
int t2_1_100 = rt2_1_100 * (4 + t2_0_201);
t2_1_100 += (1 - rt2_1_100) * t2_0_200;

int rt2_1_101 = (input[i] > (811+offset2));
int t2_1_101 = rt2_1_101 * (4 + t2_0_203);
t2_1_101 += (1 - rt2_1_101) * t2_0_202;

int rt2_1_102 = (input[i] > (819+offset2));
int t2_1_102 = rt2_1_102 * (4 + t2_0_205);
t2_1_102 += (1 - rt2_1_102) * t2_0_204;

int rt2_1_103 = (input[i] > (827+offset2));
int t2_1_103 = rt2_1_103 * (4 + t2_0_207);
t2_1_103 += (1 - rt2_1_103) * t2_0_206;

int rt2_1_104 = (input[i] > (835+offset2));
int t2_1_104 = rt2_1_104 * (4 + t2_0_209);
t2_1_104 += (1 - rt2_1_104) * t2_0_208;

int rt2_1_105 = (input[i] > (843+offset2));
int t2_1_105 = rt2_1_105 * (4 + t2_0_211);
t2_1_105 += (1 - rt2_1_105) * t2_0_210;

int rt2_1_106 = (input[i] > (851+offset2));
int t2_1_106 = rt2_1_106 * (4 + t2_0_213);
t2_1_106 += (1 - rt2_1_106) * t2_0_212;

int rt2_1_107 = (input[i] > (859+offset2));
int t2_1_107 = rt2_1_107 * (4 + t2_0_215);
t2_1_107 += (1 - rt2_1_107) * t2_0_214;

int rt2_1_108 = (input[i] > (867+offset2));
int t2_1_108 = rt2_1_108 * (4 + t2_0_217);
t2_1_108 += (1 - rt2_1_108) * t2_0_216;

int rt2_1_109 = (input[i] > (875+offset2));
int t2_1_109 = rt2_1_109 * (4 + t2_0_219);
t2_1_109 += (1 - rt2_1_109) * t2_0_218;

int rt2_1_110 = (input[i] > (883+offset2));
int t2_1_110 = rt2_1_110 * (4 + t2_0_221);
t2_1_110 += (1 - rt2_1_110) * t2_0_220;

int rt2_1_111 = (input[i] > (891+offset2));
int t2_1_111 = rt2_1_111 * (4 + t2_0_223);
t2_1_111 += (1 - rt2_1_111) * t2_0_222;

int rt2_1_112 = (input[i] > (899+offset2));
int t2_1_112 = rt2_1_112 * (4 + t2_0_225);
t2_1_112 += (1 - rt2_1_112) * t2_0_224;

int rt2_1_113 = (input[i] > (907+offset2));
int t2_1_113 = rt2_1_113 * (4 + t2_0_227);
t2_1_113 += (1 - rt2_1_113) * t2_0_226;

int rt2_1_114 = (input[i] > (915+offset2));
int t2_1_114 = rt2_1_114 * (4 + t2_0_229);
t2_1_114 += (1 - rt2_1_114) * t2_0_228;

int rt2_1_115 = (input[i] > (923+offset2));
int t2_1_115 = rt2_1_115 * (4 + t2_0_231);
t2_1_115 += (1 - rt2_1_115) * t2_0_230;

int rt2_1_116 = (input[i] > (931+offset2));
int t2_1_116 = rt2_1_116 * (4 + t2_0_233);
t2_1_116 += (1 - rt2_1_116) * t2_0_232;

int rt2_1_117 = (input[i] > (939+offset2));
int t2_1_117 = rt2_1_117 * (4 + t2_0_235);
t2_1_117 += (1 - rt2_1_117) * t2_0_234;

int rt2_1_118 = (input[i] > (947+offset2));
int t2_1_118 = rt2_1_118 * (4 + t2_0_237);
t2_1_118 += (1 - rt2_1_118) * t2_0_236;

int rt2_1_119 = (input[i] > (955+offset2));
int t2_1_119 = rt2_1_119 * (4 + t2_0_239);
t2_1_119 += (1 - rt2_1_119) * t2_0_238;

int rt2_1_120 = (input[i] > (963+offset2));
int t2_1_120 = rt2_1_120 * (4 + t2_0_241);
t2_1_120 += (1 - rt2_1_120) * t2_0_240;

int rt2_1_121 = (input[i] > (971+offset2));
int t2_1_121 = rt2_1_121 * (4 + t2_0_243);
t2_1_121 += (1 - rt2_1_121) * t2_0_242;

int rt2_1_122 = (input[i] > (979+offset2));
int t2_1_122 = rt2_1_122 * (4 + t2_0_245);
t2_1_122 += (1 - rt2_1_122) * t2_0_244;

int rt2_1_123 = (input[i] > (987+offset2));
int t2_1_123 = rt2_1_123 * (4 + t2_0_247);
t2_1_123 += (1 - rt2_1_123) * t2_0_246;

int rt2_1_124 = (input[i] > (995+offset2));
int t2_1_124 = rt2_1_124 * (4 + t2_0_249);
t2_1_124 += (1 - rt2_1_124) * t2_0_248;

int rt2_1_125 = (input[i] > (1003+offset2));
int t2_1_125 = rt2_1_125 * (4 + t2_0_251);
t2_1_125 += (1 - rt2_1_125) * t2_0_250;

int rt2_1_126 = (input[i] > (1011+offset2));
int t2_1_126 = rt2_1_126 * (4 + t2_0_253);
t2_1_126 += (1 - rt2_1_126) * t2_0_252;

int rt2_1_127 = (input[i] > (1019+offset2));
int t2_1_127 = rt2_1_127 * (4 + t2_0_255);
t2_1_127 += (1 - rt2_1_127) * t2_0_254;

int rt2_2_0 = (input[i] > (7+offset2));
int t2_2_0 = rt2_2_0 * (8 + t2_1_1);
t2_2_0 += (1 - rt2_2_0) * t2_1_0;

int rt2_2_1 = (input[i] > (23+offset2));
int t2_2_1 = rt2_2_1 * (8 + t2_1_3);
t2_2_1 += (1 - rt2_2_1) * t2_1_2;

int rt2_2_2 = (input[i] > (39+offset2));
int t2_2_2 = rt2_2_2 * (8 + t2_1_5);
t2_2_2 += (1 - rt2_2_2) * t2_1_4;

int rt2_2_3 = (input[i] > (55+offset2));
int t2_2_3 = rt2_2_3 * (8 + t2_1_7);
t2_2_3 += (1 - rt2_2_3) * t2_1_6;

int rt2_2_4 = (input[i] > (71+offset2));
int t2_2_4 = rt2_2_4 * (8 + t2_1_9);
t2_2_4 += (1 - rt2_2_4) * t2_1_8;

int rt2_2_5 = (input[i] > (87+offset2));
int t2_2_5 = rt2_2_5 * (8 + t2_1_11);
t2_2_5 += (1 - rt2_2_5) * t2_1_10;

int rt2_2_6 = (input[i] > (103+offset2));
int t2_2_6 = rt2_2_6 * (8 + t2_1_13);
t2_2_6 += (1 - rt2_2_6) * t2_1_12;

int rt2_2_7 = (input[i] > (119+offset2));
int t2_2_7 = rt2_2_7 * (8 + t2_1_15);
t2_2_7 += (1 - rt2_2_7) * t2_1_14;

int rt2_2_8 = (input[i] > (135+offset2));
int t2_2_8 = rt2_2_8 * (8 + t2_1_17);
t2_2_8 += (1 - rt2_2_8) * t2_1_16;

int rt2_2_9 = (input[i] > (151+offset2));
int t2_2_9 = rt2_2_9 * (8 + t2_1_19);
t2_2_9 += (1 - rt2_2_9) * t2_1_18;

int rt2_2_10 = (input[i] > (167+offset2));
int t2_2_10 = rt2_2_10 * (8 + t2_1_21);
t2_2_10 += (1 - rt2_2_10) * t2_1_20;

int rt2_2_11 = (input[i] > (183+offset2));
int t2_2_11 = rt2_2_11 * (8 + t2_1_23);
t2_2_11 += (1 - rt2_2_11) * t2_1_22;

int rt2_2_12 = (input[i] > (199+offset2));
int t2_2_12 = rt2_2_12 * (8 + t2_1_25);
t2_2_12 += (1 - rt2_2_12) * t2_1_24;

int rt2_2_13 = (input[i] > (215+offset2));
int t2_2_13 = rt2_2_13 * (8 + t2_1_27);
t2_2_13 += (1 - rt2_2_13) * t2_1_26;

int rt2_2_14 = (input[i] > (231+offset2));
int t2_2_14 = rt2_2_14 * (8 + t2_1_29);
t2_2_14 += (1 - rt2_2_14) * t2_1_28;

int rt2_2_15 = (input[i] > (247+offset2));
int t2_2_15 = rt2_2_15 * (8 + t2_1_31);
t2_2_15 += (1 - rt2_2_15) * t2_1_30;

int rt2_2_16 = (input[i] > (263+offset2));
int t2_2_16 = rt2_2_16 * (8 + t2_1_33);
t2_2_16 += (1 - rt2_2_16) * t2_1_32;

int rt2_2_17 = (input[i] > (279+offset2));
int t2_2_17 = rt2_2_17 * (8 + t2_1_35);
t2_2_17 += (1 - rt2_2_17) * t2_1_34;

int rt2_2_18 = (input[i] > (295+offset2));
int t2_2_18 = rt2_2_18 * (8 + t2_1_37);
t2_2_18 += (1 - rt2_2_18) * t2_1_36;

int rt2_2_19 = (input[i] > (311+offset2));
int t2_2_19 = rt2_2_19 * (8 + t2_1_39);
t2_2_19 += (1 - rt2_2_19) * t2_1_38;

int rt2_2_20 = (input[i] > (327+offset2));
int t2_2_20 = rt2_2_20 * (8 + t2_1_41);
t2_2_20 += (1 - rt2_2_20) * t2_1_40;

int rt2_2_21 = (input[i] > (343+offset2));
int t2_2_21 = rt2_2_21 * (8 + t2_1_43);
t2_2_21 += (1 - rt2_2_21) * t2_1_42;

int rt2_2_22 = (input[i] > (359+offset2));
int t2_2_22 = rt2_2_22 * (8 + t2_1_45);
t2_2_22 += (1 - rt2_2_22) * t2_1_44;

int rt2_2_23 = (input[i] > (375+offset2));
int t2_2_23 = rt2_2_23 * (8 + t2_1_47);
t2_2_23 += (1 - rt2_2_23) * t2_1_46;

int rt2_2_24 = (input[i] > (391+offset2));
int t2_2_24 = rt2_2_24 * (8 + t2_1_49);
t2_2_24 += (1 - rt2_2_24) * t2_1_48;

int rt2_2_25 = (input[i] > (407+offset2));
int t2_2_25 = rt2_2_25 * (8 + t2_1_51);
t2_2_25 += (1 - rt2_2_25) * t2_1_50;

int rt2_2_26 = (input[i] > (423+offset2));
int t2_2_26 = rt2_2_26 * (8 + t2_1_53);
t2_2_26 += (1 - rt2_2_26) * t2_1_52;

int rt2_2_27 = (input[i] > (439+offset2));
int t2_2_27 = rt2_2_27 * (8 + t2_1_55);
t2_2_27 += (1 - rt2_2_27) * t2_1_54;

int rt2_2_28 = (input[i] > (455+offset2));
int t2_2_28 = rt2_2_28 * (8 + t2_1_57);
t2_2_28 += (1 - rt2_2_28) * t2_1_56;

int rt2_2_29 = (input[i] > (471+offset2));
int t2_2_29 = rt2_2_29 * (8 + t2_1_59);
t2_2_29 += (1 - rt2_2_29) * t2_1_58;

int rt2_2_30 = (input[i] > (487+offset2));
int t2_2_30 = rt2_2_30 * (8 + t2_1_61);
t2_2_30 += (1 - rt2_2_30) * t2_1_60;

int rt2_2_31 = (input[i] > (503+offset2));
int t2_2_31 = rt2_2_31 * (8 + t2_1_63);
t2_2_31 += (1 - rt2_2_31) * t2_1_62;

int rt2_2_32 = (input[i] > (519+offset2));
int t2_2_32 = rt2_2_32 * (8 + t2_1_65);
t2_2_32 += (1 - rt2_2_32) * t2_1_64;

int rt2_2_33 = (input[i] > (535+offset2));
int t2_2_33 = rt2_2_33 * (8 + t2_1_67);
t2_2_33 += (1 - rt2_2_33) * t2_1_66;

int rt2_2_34 = (input[i] > (551+offset2));
int t2_2_34 = rt2_2_34 * (8 + t2_1_69);
t2_2_34 += (1 - rt2_2_34) * t2_1_68;

int rt2_2_35 = (input[i] > (567+offset2));
int t2_2_35 = rt2_2_35 * (8 + t2_1_71);
t2_2_35 += (1 - rt2_2_35) * t2_1_70;

int rt2_2_36 = (input[i] > (583+offset2));
int t2_2_36 = rt2_2_36 * (8 + t2_1_73);
t2_2_36 += (1 - rt2_2_36) * t2_1_72;

int rt2_2_37 = (input[i] > (599+offset2));
int t2_2_37 = rt2_2_37 * (8 + t2_1_75);
t2_2_37 += (1 - rt2_2_37) * t2_1_74;

int rt2_2_38 = (input[i] > (615+offset2));
int t2_2_38 = rt2_2_38 * (8 + t2_1_77);
t2_2_38 += (1 - rt2_2_38) * t2_1_76;

int rt2_2_39 = (input[i] > (631+offset2));
int t2_2_39 = rt2_2_39 * (8 + t2_1_79);
t2_2_39 += (1 - rt2_2_39) * t2_1_78;

int rt2_2_40 = (input[i] > (647+offset2));
int t2_2_40 = rt2_2_40 * (8 + t2_1_81);
t2_2_40 += (1 - rt2_2_40) * t2_1_80;

int rt2_2_41 = (input[i] > (663+offset2));
int t2_2_41 = rt2_2_41 * (8 + t2_1_83);
t2_2_41 += (1 - rt2_2_41) * t2_1_82;

int rt2_2_42 = (input[i] > (679+offset2));
int t2_2_42 = rt2_2_42 * (8 + t2_1_85);
t2_2_42 += (1 - rt2_2_42) * t2_1_84;

int rt2_2_43 = (input[i] > (695+offset2));
int t2_2_43 = rt2_2_43 * (8 + t2_1_87);
t2_2_43 += (1 - rt2_2_43) * t2_1_86;

int rt2_2_44 = (input[i] > (711+offset2));
int t2_2_44 = rt2_2_44 * (8 + t2_1_89);
t2_2_44 += (1 - rt2_2_44) * t2_1_88;

int rt2_2_45 = (input[i] > (727+offset2));
int t2_2_45 = rt2_2_45 * (8 + t2_1_91);
t2_2_45 += (1 - rt2_2_45) * t2_1_90;

int rt2_2_46 = (input[i] > (743+offset2));
int t2_2_46 = rt2_2_46 * (8 + t2_1_93);
t2_2_46 += (1 - rt2_2_46) * t2_1_92;

int rt2_2_47 = (input[i] > (759+offset2));
int t2_2_47 = rt2_2_47 * (8 + t2_1_95);
t2_2_47 += (1 - rt2_2_47) * t2_1_94;

int rt2_2_48 = (input[i] > (775+offset2));
int t2_2_48 = rt2_2_48 * (8 + t2_1_97);
t2_2_48 += (1 - rt2_2_48) * t2_1_96;

int rt2_2_49 = (input[i] > (791+offset2));
int t2_2_49 = rt2_2_49 * (8 + t2_1_99);
t2_2_49 += (1 - rt2_2_49) * t2_1_98;

int rt2_2_50 = (input[i] > (807+offset2));
int t2_2_50 = rt2_2_50 * (8 + t2_1_101);
t2_2_50 += (1 - rt2_2_50) * t2_1_100;

int rt2_2_51 = (input[i] > (823+offset2));
int t2_2_51 = rt2_2_51 * (8 + t2_1_103);
t2_2_51 += (1 - rt2_2_51) * t2_1_102;

int rt2_2_52 = (input[i] > (839+offset2));
int t2_2_52 = rt2_2_52 * (8 + t2_1_105);
t2_2_52 += (1 - rt2_2_52) * t2_1_104;

int rt2_2_53 = (input[i] > (855+offset2));
int t2_2_53 = rt2_2_53 * (8 + t2_1_107);
t2_2_53 += (1 - rt2_2_53) * t2_1_106;

int rt2_2_54 = (input[i] > (871+offset2));
int t2_2_54 = rt2_2_54 * (8 + t2_1_109);
t2_2_54 += (1 - rt2_2_54) * t2_1_108;

int rt2_2_55 = (input[i] > (887+offset2));
int t2_2_55 = rt2_2_55 * (8 + t2_1_111);
t2_2_55 += (1 - rt2_2_55) * t2_1_110;

int rt2_2_56 = (input[i] > (903+offset2));
int t2_2_56 = rt2_2_56 * (8 + t2_1_113);
t2_2_56 += (1 - rt2_2_56) * t2_1_112;

int rt2_2_57 = (input[i] > (919+offset2));
int t2_2_57 = rt2_2_57 * (8 + t2_1_115);
t2_2_57 += (1 - rt2_2_57) * t2_1_114;

int rt2_2_58 = (input[i] > (935+offset2));
int t2_2_58 = rt2_2_58 * (8 + t2_1_117);
t2_2_58 += (1 - rt2_2_58) * t2_1_116;

int rt2_2_59 = (input[i] > (951+offset2));
int t2_2_59 = rt2_2_59 * (8 + t2_1_119);
t2_2_59 += (1 - rt2_2_59) * t2_1_118;

int rt2_2_60 = (input[i] > (967+offset2));
int t2_2_60 = rt2_2_60 * (8 + t2_1_121);
t2_2_60 += (1 - rt2_2_60) * t2_1_120;

int rt2_2_61 = (input[i] > (983+offset2));
int t2_2_61 = rt2_2_61 * (8 + t2_1_123);
t2_2_61 += (1 - rt2_2_61) * t2_1_122;

int rt2_2_62 = (input[i] > (999+offset2));
int t2_2_62 = rt2_2_62 * (8 + t2_1_125);
t2_2_62 += (1 - rt2_2_62) * t2_1_124;

int rt2_2_63 = (input[i] > (1015+offset2));
int t2_2_63 = rt2_2_63 * (8 + t2_1_127);
t2_2_63 += (1 - rt2_2_63) * t2_1_126;

int rt2_3_0 = (input[i] > (15+offset2));
int t2_3_0 = rt2_3_0 * (16 + t2_2_1);
t2_3_0 += (1 - rt2_3_0) * t2_2_0;

int rt2_3_1 = (input[i] > (47+offset2));
int t2_3_1 = rt2_3_1 * (16 + t2_2_3);
t2_3_1 += (1 - rt2_3_1) * t2_2_2;

int rt2_3_2 = (input[i] > (79+offset2));
int t2_3_2 = rt2_3_2 * (16 + t2_2_5);
t2_3_2 += (1 - rt2_3_2) * t2_2_4;

int rt2_3_3 = (input[i] > (111+offset2));
int t2_3_3 = rt2_3_3 * (16 + t2_2_7);
t2_3_3 += (1 - rt2_3_3) * t2_2_6;

int rt2_3_4 = (input[i] > (143+offset2));
int t2_3_4 = rt2_3_4 * (16 + t2_2_9);
t2_3_4 += (1 - rt2_3_4) * t2_2_8;

int rt2_3_5 = (input[i] > (175+offset2));
int t2_3_5 = rt2_3_5 * (16 + t2_2_11);
t2_3_5 += (1 - rt2_3_5) * t2_2_10;

int rt2_3_6 = (input[i] > (207+offset2));
int t2_3_6 = rt2_3_6 * (16 + t2_2_13);
t2_3_6 += (1 - rt2_3_6) * t2_2_12;

int rt2_3_7 = (input[i] > (239+offset2));
int t2_3_7 = rt2_3_7 * (16 + t2_2_15);
t2_3_7 += (1 - rt2_3_7) * t2_2_14;

int rt2_3_8 = (input[i] > (271+offset2));
int t2_3_8 = rt2_3_8 * (16 + t2_2_17);
t2_3_8 += (1 - rt2_3_8) * t2_2_16;

int rt2_3_9 = (input[i] > (303+offset2));
int t2_3_9 = rt2_3_9 * (16 + t2_2_19);
t2_3_9 += (1 - rt2_3_9) * t2_2_18;

int rt2_3_10 = (input[i] > (335+offset2));
int t2_3_10 = rt2_3_10 * (16 + t2_2_21);
t2_3_10 += (1 - rt2_3_10) * t2_2_20;

int rt2_3_11 = (input[i] > (367+offset2));
int t2_3_11 = rt2_3_11 * (16 + t2_2_23);
t2_3_11 += (1 - rt2_3_11) * t2_2_22;

int rt2_3_12 = (input[i] > (399+offset2));
int t2_3_12 = rt2_3_12 * (16 + t2_2_25);
t2_3_12 += (1 - rt2_3_12) * t2_2_24;

int rt2_3_13 = (input[i] > (431+offset2));
int t2_3_13 = rt2_3_13 * (16 + t2_2_27);
t2_3_13 += (1 - rt2_3_13) * t2_2_26;

int rt2_3_14 = (input[i] > (463+offset2));
int t2_3_14 = rt2_3_14 * (16 + t2_2_29);
t2_3_14 += (1 - rt2_3_14) * t2_2_28;

int rt2_3_15 = (input[i] > (495+offset2));
int t2_3_15 = rt2_3_15 * (16 + t2_2_31);
t2_3_15 += (1 - rt2_3_15) * t2_2_30;

int rt2_3_16 = (input[i] > (527+offset2));
int t2_3_16 = rt2_3_16 * (16 + t2_2_33);
t2_3_16 += (1 - rt2_3_16) * t2_2_32;

int rt2_3_17 = (input[i] > (559+offset2));
int t2_3_17 = rt2_3_17 * (16 + t2_2_35);
t2_3_17 += (1 - rt2_3_17) * t2_2_34;

int rt2_3_18 = (input[i] > (591+offset2));
int t2_3_18 = rt2_3_18 * (16 + t2_2_37);
t2_3_18 += (1 - rt2_3_18) * t2_2_36;

int rt2_3_19 = (input[i] > (623+offset2));
int t2_3_19 = rt2_3_19 * (16 + t2_2_39);
t2_3_19 += (1 - rt2_3_19) * t2_2_38;

int rt2_3_20 = (input[i] > (655+offset2));
int t2_3_20 = rt2_3_20 * (16 + t2_2_41);
t2_3_20 += (1 - rt2_3_20) * t2_2_40;

int rt2_3_21 = (input[i] > (687+offset2));
int t2_3_21 = rt2_3_21 * (16 + t2_2_43);
t2_3_21 += (1 - rt2_3_21) * t2_2_42;

int rt2_3_22 = (input[i] > (719+offset2));
int t2_3_22 = rt2_3_22 * (16 + t2_2_45);
t2_3_22 += (1 - rt2_3_22) * t2_2_44;

int rt2_3_23 = (input[i] > (751+offset2));
int t2_3_23 = rt2_3_23 * (16 + t2_2_47);
t2_3_23 += (1 - rt2_3_23) * t2_2_46;

int rt2_3_24 = (input[i] > (783+offset2));
int t2_3_24 = rt2_3_24 * (16 + t2_2_49);
t2_3_24 += (1 - rt2_3_24) * t2_2_48;

int rt2_3_25 = (input[i] > (815+offset2));
int t2_3_25 = rt2_3_25 * (16 + t2_2_51);
t2_3_25 += (1 - rt2_3_25) * t2_2_50;

int rt2_3_26 = (input[i] > (847+offset2));
int t2_3_26 = rt2_3_26 * (16 + t2_2_53);
t2_3_26 += (1 - rt2_3_26) * t2_2_52;

int rt2_3_27 = (input[i] > (879+offset2));
int t2_3_27 = rt2_3_27 * (16 + t2_2_55);
t2_3_27 += (1 - rt2_3_27) * t2_2_54;

int rt2_3_28 = (input[i] > (911+offset2));
int t2_3_28 = rt2_3_28 * (16 + t2_2_57);
t2_3_28 += (1 - rt2_3_28) * t2_2_56;

int rt2_3_29 = (input[i] > (943+offset2));
int t2_3_29 = rt2_3_29 * (16 + t2_2_59);
t2_3_29 += (1 - rt2_3_29) * t2_2_58;

int rt2_3_30 = (input[i] > (975+offset2));
int t2_3_30 = rt2_3_30 * (16 + t2_2_61);
t2_3_30 += (1 - rt2_3_30) * t2_2_60;

int rt2_3_31 = (input[i] > (1007+offset2));
int t2_3_31 = rt2_3_31 * (16 + t2_2_63);
t2_3_31 += (1 - rt2_3_31) * t2_2_62;

int rt2_4_0 = (input[i] > (31+offset2));
int t2_4_0 = rt2_4_0 * (32 + t2_3_1);
t2_4_0 += (1 - rt2_4_0) * t2_3_0;

int rt2_4_1 = (input[i] > (95+offset2));
int t2_4_1 = rt2_4_1 * (32 + t2_3_3);
t2_4_1 += (1 - rt2_4_1) * t2_3_2;

int rt2_4_2 = (input[i] > (159+offset2));
int t2_4_2 = rt2_4_2 * (32 + t2_3_5);
t2_4_2 += (1 - rt2_4_2) * t2_3_4;

int rt2_4_3 = (input[i] > (223+offset2));
int t2_4_3 = rt2_4_3 * (32 + t2_3_7);
t2_4_3 += (1 - rt2_4_3) * t2_3_6;

int rt2_4_4 = (input[i] > (287+offset2));
int t2_4_4 = rt2_4_4 * (32 + t2_3_9);
t2_4_4 += (1 - rt2_4_4) * t2_3_8;

int rt2_4_5 = (input[i] > (351+offset2));
int t2_4_5 = rt2_4_5 * (32 + t2_3_11);
t2_4_5 += (1 - rt2_4_5) * t2_3_10;

int rt2_4_6 = (input[i] > (415+offset2));
int t2_4_6 = rt2_4_6 * (32 + t2_3_13);
t2_4_6 += (1 - rt2_4_6) * t2_3_12;

int rt2_4_7 = (input[i] > (479+offset2));
int t2_4_7 = rt2_4_7 * (32 + t2_3_15);
t2_4_7 += (1 - rt2_4_7) * t2_3_14;

int rt2_4_8 = (input[i] > (543+offset2));
int t2_4_8 = rt2_4_8 * (32 + t2_3_17);
t2_4_8 += (1 - rt2_4_8) * t2_3_16;

int rt2_4_9 = (input[i] > (607+offset2));
int t2_4_9 = rt2_4_9 * (32 + t2_3_19);
t2_4_9 += (1 - rt2_4_9) * t2_3_18;

int rt2_4_10 = (input[i] > (671+offset2));
int t2_4_10 = rt2_4_10 * (32 + t2_3_21);
t2_4_10 += (1 - rt2_4_10) * t2_3_20;

int rt2_4_11 = (input[i] > (735+offset2));
int t2_4_11 = rt2_4_11 * (32 + t2_3_23);
t2_4_11 += (1 - rt2_4_11) * t2_3_22;

int rt2_4_12 = (input[i] > (799+offset2));
int t2_4_12 = rt2_4_12 * (32 + t2_3_25);
t2_4_12 += (1 - rt2_4_12) * t2_3_24;

int rt2_4_13 = (input[i] > (863+offset2));
int t2_4_13 = rt2_4_13 * (32 + t2_3_27);
t2_4_13 += (1 - rt2_4_13) * t2_3_26;

int rt2_4_14 = (input[i] > (927+offset2));
int t2_4_14 = rt2_4_14 * (32 + t2_3_29);
t2_4_14 += (1 - rt2_4_14) * t2_3_28;

int rt2_4_15 = (input[i] > (991+offset2));
int t2_4_15 = rt2_4_15 * (32 + t2_3_31);
t2_4_15 += (1 - rt2_4_15) * t2_3_30;

int rt2_5_0 = (input[i] > (63+offset2));
int t2_5_0 = rt2_5_0 * (64 + t2_4_1);
t2_5_0 += (1 - rt2_5_0) * t2_4_0;

int rt2_5_1 = (input[i] > (191+offset2));
int t2_5_1 = rt2_5_1 * (64 + t2_4_3);
t2_5_1 += (1 - rt2_5_1) * t2_4_2;

int rt2_5_2 = (input[i] > (319+offset2));
int t2_5_2 = rt2_5_2 * (64 + t2_4_5);
t2_5_2 += (1 - rt2_5_2) * t2_4_4;

int rt2_5_3 = (input[i] > (447+offset2));
int t2_5_3 = rt2_5_3 * (64 + t2_4_7);
t2_5_3 += (1 - rt2_5_3) * t2_4_6;

int rt2_5_4 = (input[i] > (575+offset2));
int t2_5_4 = rt2_5_4 * (64 + t2_4_9);
t2_5_4 += (1 - rt2_5_4) * t2_4_8;

int rt2_5_5 = (input[i] > (703+offset2));
int t2_5_5 = rt2_5_5 * (64 + t2_4_11);
t2_5_5 += (1 - rt2_5_5) * t2_4_10;

int rt2_5_6 = (input[i] > (831+offset2));
int t2_5_6 = rt2_5_6 * (64 + t2_4_13);
t2_5_6 += (1 - rt2_5_6) * t2_4_12;

int rt2_5_7 = (input[i] > (959+offset2));
int t2_5_7 = rt2_5_7 * (64 + t2_4_15);
t2_5_7 += (1 - rt2_5_7) * t2_4_14;

int rt2_6_0 = (input[i] > (127+offset2));
int t2_6_0 = rt2_6_0 * (128 + t2_5_1);
t2_6_0 += (1 - rt2_6_0) * t2_5_0;

int rt2_6_1 = (input[i] > (383+offset2));
int t2_6_1 = rt2_6_1 * (128 + t2_5_3);
t2_6_1 += (1 - rt2_6_1) * t2_5_2;

int rt2_6_2 = (input[i] > (639+offset2));
int t2_6_2 = rt2_6_2 * (128 + t2_5_5);
t2_6_2 += (1 - rt2_6_2) * t2_5_4;

int rt2_6_3 = (input[i] > (895+offset2));
int t2_6_3 = rt2_6_3 * (128 + t2_5_7);
t2_6_3 += (1 - rt2_6_3) * t2_5_6;

int rt2_7_0 = (input[i] > (255+offset2));
int t2_7_0 = rt2_7_0 * (256 + t2_6_1);
t2_7_0 += (1 - rt2_7_0) * t2_6_0;

int rt2_7_1 = (input[i] > (767+offset2));
int t2_7_1 = rt2_7_1 * (256 + t2_6_3);
t2_7_1 += (1 - rt2_7_1) * t2_6_2;

int rt2_8_0 = (input[i] > (511+offset2));
int t2_8_0 = rt2_8_0 * (512 + t2_7_1);
t2_8_0 += (1 - rt2_8_0) * t2_7_0;

int rt3_0_0 = (input[i] > (1+offset3));
int t3_0_0 = rt3_0_0 * (2 + (input[i] > (2+offset3)));
t3_0_0 += (1 - rt3_0_0) * (input[i] > (0+offset3));

int rt3_0_1 = (input[i] > (5+offset3));
int t3_0_1 = rt3_0_1 * (2 + (input[i] > (6+offset3)));
t3_0_1 += (1 - rt3_0_1) * (input[i] > (4+offset3));

int rt3_0_2 = (input[i] > (9+offset3));
int t3_0_2 = rt3_0_2 * (2 + (input[i] > (10+offset3)));
t3_0_2 += (1 - rt3_0_2) * (input[i] > (8+offset3));

int rt3_0_3 = (input[i] > (13+offset3));
int t3_0_3 = rt3_0_3 * (2 + (input[i] > (14+offset3)));
t3_0_3 += (1 - rt3_0_3) * (input[i] > (12+offset3));

int rt3_0_4 = (input[i] > (17+offset3));
int t3_0_4 = rt3_0_4 * (2 + (input[i] > (18+offset3)));
t3_0_4 += (1 - rt3_0_4) * (input[i] > (16+offset3));

int rt3_0_5 = (input[i] > (21+offset3));
int t3_0_5 = rt3_0_5 * (2 + (input[i] > (22+offset3)));
t3_0_5 += (1 - rt3_0_5) * (input[i] > (20+offset3));

int rt3_0_6 = (input[i] > (25+offset3));
int t3_0_6 = rt3_0_6 * (2 + (input[i] > (26+offset3)));
t3_0_6 += (1 - rt3_0_6) * (input[i] > (24+offset3));

int rt3_0_7 = (input[i] > (29+offset3));
int t3_0_7 = rt3_0_7 * (2 + (input[i] > (30+offset3)));
t3_0_7 += (1 - rt3_0_7) * (input[i] > (28+offset3));

int rt3_0_8 = (input[i] > (33+offset3));
int t3_0_8 = rt3_0_8 * (2 + (input[i] > (34+offset3)));
t3_0_8 += (1 - rt3_0_8) * (input[i] > (32+offset3));

int rt3_0_9 = (input[i] > (37+offset3));
int t3_0_9 = rt3_0_9 * (2 + (input[i] > (38+offset3)));
t3_0_9 += (1 - rt3_0_9) * (input[i] > (36+offset3));

int rt3_0_10 = (input[i] > (41+offset3));
int t3_0_10 = rt3_0_10 * (2 + (input[i] > (42+offset3)));
t3_0_10 += (1 - rt3_0_10) * (input[i] > (40+offset3));

int rt3_0_11 = (input[i] > (45+offset3));
int t3_0_11 = rt3_0_11 * (2 + (input[i] > (46+offset3)));
t3_0_11 += (1 - rt3_0_11) * (input[i] > (44+offset3));

int rt3_0_12 = (input[i] > (49+offset3));
int t3_0_12 = rt3_0_12 * (2 + (input[i] > (50+offset3)));
t3_0_12 += (1 - rt3_0_12) * (input[i] > (48+offset3));

int rt3_0_13 = (input[i] > (53+offset3));
int t3_0_13 = rt3_0_13 * (2 + (input[i] > (54+offset3)));
t3_0_13 += (1 - rt3_0_13) * (input[i] > (52+offset3));

int rt3_0_14 = (input[i] > (57+offset3));
int t3_0_14 = rt3_0_14 * (2 + (input[i] > (58+offset3)));
t3_0_14 += (1 - rt3_0_14) * (input[i] > (56+offset3));

int rt3_0_15 = (input[i] > (61+offset3));
int t3_0_15 = rt3_0_15 * (2 + (input[i] > (62+offset3)));
t3_0_15 += (1 - rt3_0_15) * (input[i] > (60+offset3));

int rt3_0_16 = (input[i] > (65+offset3));
int t3_0_16 = rt3_0_16 * (2 + (input[i] > (66+offset3)));
t3_0_16 += (1 - rt3_0_16) * (input[i] > (64+offset3));

int rt3_0_17 = (input[i] > (69+offset3));
int t3_0_17 = rt3_0_17 * (2 + (input[i] > (70+offset3)));
t3_0_17 += (1 - rt3_0_17) * (input[i] > (68+offset3));

int rt3_0_18 = (input[i] > (73+offset3));
int t3_0_18 = rt3_0_18 * (2 + (input[i] > (74+offset3)));
t3_0_18 += (1 - rt3_0_18) * (input[i] > (72+offset3));

int rt3_0_19 = (input[i] > (77+offset3));
int t3_0_19 = rt3_0_19 * (2 + (input[i] > (78+offset3)));
t3_0_19 += (1 - rt3_0_19) * (input[i] > (76+offset3));

int rt3_0_20 = (input[i] > (81+offset3));
int t3_0_20 = rt3_0_20 * (2 + (input[i] > (82+offset3)));
t3_0_20 += (1 - rt3_0_20) * (input[i] > (80+offset3));

int rt3_0_21 = (input[i] > (85+offset3));
int t3_0_21 = rt3_0_21 * (2 + (input[i] > (86+offset3)));
t3_0_21 += (1 - rt3_0_21) * (input[i] > (84+offset3));

int rt3_0_22 = (input[i] > (89+offset3));
int t3_0_22 = rt3_0_22 * (2 + (input[i] > (90+offset3)));
t3_0_22 += (1 - rt3_0_22) * (input[i] > (88+offset3));

int rt3_0_23 = (input[i] > (93+offset3));
int t3_0_23 = rt3_0_23 * (2 + (input[i] > (94+offset3)));
t3_0_23 += (1 - rt3_0_23) * (input[i] > (92+offset3));

int rt3_0_24 = (input[i] > (97+offset3));
int t3_0_24 = rt3_0_24 * (2 + (input[i] > (98+offset3)));
t3_0_24 += (1 - rt3_0_24) * (input[i] > (96+offset3));

int rt3_0_25 = (input[i] > (101+offset3));
int t3_0_25 = rt3_0_25 * (2 + (input[i] > (102+offset3)));
t3_0_25 += (1 - rt3_0_25) * (input[i] > (100+offset3));

int rt3_0_26 = (input[i] > (105+offset3));
int t3_0_26 = rt3_0_26 * (2 + (input[i] > (106+offset3)));
t3_0_26 += (1 - rt3_0_26) * (input[i] > (104+offset3));

int rt3_0_27 = (input[i] > (109+offset3));
int t3_0_27 = rt3_0_27 * (2 + (input[i] > (110+offset3)));
t3_0_27 += (1 - rt3_0_27) * (input[i] > (108+offset3));

int rt3_0_28 = (input[i] > (113+offset3));
int t3_0_28 = rt3_0_28 * (2 + (input[i] > (114+offset3)));
t3_0_28 += (1 - rt3_0_28) * (input[i] > (112+offset3));

int rt3_0_29 = (input[i] > (117+offset3));
int t3_0_29 = rt3_0_29 * (2 + (input[i] > (118+offset3)));
t3_0_29 += (1 - rt3_0_29) * (input[i] > (116+offset3));

int rt3_0_30 = (input[i] > (121+offset3));
int t3_0_30 = rt3_0_30 * (2 + (input[i] > (122+offset3)));
t3_0_30 += (1 - rt3_0_30) * (input[i] > (120+offset3));

int rt3_0_31 = (input[i] > (125+offset3));
int t3_0_31 = rt3_0_31 * (2 + (input[i] > (126+offset3)));
t3_0_31 += (1 - rt3_0_31) * (input[i] > (124+offset3));

int rt3_0_32 = (input[i] > (129+offset3));
int t3_0_32 = rt3_0_32 * (2 + (input[i] > (130+offset3)));
t3_0_32 += (1 - rt3_0_32) * (input[i] > (128+offset3));

int rt3_0_33 = (input[i] > (133+offset3));
int t3_0_33 = rt3_0_33 * (2 + (input[i] > (134+offset3)));
t3_0_33 += (1 - rt3_0_33) * (input[i] > (132+offset3));

int rt3_0_34 = (input[i] > (137+offset3));
int t3_0_34 = rt3_0_34 * (2 + (input[i] > (138+offset3)));
t3_0_34 += (1 - rt3_0_34) * (input[i] > (136+offset3));

int rt3_0_35 = (input[i] > (141+offset3));
int t3_0_35 = rt3_0_35 * (2 + (input[i] > (142+offset3)));
t3_0_35 += (1 - rt3_0_35) * (input[i] > (140+offset3));

int rt3_0_36 = (input[i] > (145+offset3));
int t3_0_36 = rt3_0_36 * (2 + (input[i] > (146+offset3)));
t3_0_36 += (1 - rt3_0_36) * (input[i] > (144+offset3));

int rt3_0_37 = (input[i] > (149+offset3));
int t3_0_37 = rt3_0_37 * (2 + (input[i] > (150+offset3)));
t3_0_37 += (1 - rt3_0_37) * (input[i] > (148+offset3));

int rt3_0_38 = (input[i] > (153+offset3));
int t3_0_38 = rt3_0_38 * (2 + (input[i] > (154+offset3)));
t3_0_38 += (1 - rt3_0_38) * (input[i] > (152+offset3));

int rt3_0_39 = (input[i] > (157+offset3));
int t3_0_39 = rt3_0_39 * (2 + (input[i] > (158+offset3)));
t3_0_39 += (1 - rt3_0_39) * (input[i] > (156+offset3));

int rt3_0_40 = (input[i] > (161+offset3));
int t3_0_40 = rt3_0_40 * (2 + (input[i] > (162+offset3)));
t3_0_40 += (1 - rt3_0_40) * (input[i] > (160+offset3));

int rt3_0_41 = (input[i] > (165+offset3));
int t3_0_41 = rt3_0_41 * (2 + (input[i] > (166+offset3)));
t3_0_41 += (1 - rt3_0_41) * (input[i] > (164+offset3));

int rt3_0_42 = (input[i] > (169+offset3));
int t3_0_42 = rt3_0_42 * (2 + (input[i] > (170+offset3)));
t3_0_42 += (1 - rt3_0_42) * (input[i] > (168+offset3));

int rt3_0_43 = (input[i] > (173+offset3));
int t3_0_43 = rt3_0_43 * (2 + (input[i] > (174+offset3)));
t3_0_43 += (1 - rt3_0_43) * (input[i] > (172+offset3));

int rt3_0_44 = (input[i] > (177+offset3));
int t3_0_44 = rt3_0_44 * (2 + (input[i] > (178+offset3)));
t3_0_44 += (1 - rt3_0_44) * (input[i] > (176+offset3));

int rt3_0_45 = (input[i] > (181+offset3));
int t3_0_45 = rt3_0_45 * (2 + (input[i] > (182+offset3)));
t3_0_45 += (1 - rt3_0_45) * (input[i] > (180+offset3));

int rt3_0_46 = (input[i] > (185+offset3));
int t3_0_46 = rt3_0_46 * (2 + (input[i] > (186+offset3)));
t3_0_46 += (1 - rt3_0_46) * (input[i] > (184+offset3));

int rt3_0_47 = (input[i] > (189+offset3));
int t3_0_47 = rt3_0_47 * (2 + (input[i] > (190+offset3)));
t3_0_47 += (1 - rt3_0_47) * (input[i] > (188+offset3));

int rt3_0_48 = (input[i] > (193+offset3));
int t3_0_48 = rt3_0_48 * (2 + (input[i] > (194+offset3)));
t3_0_48 += (1 - rt3_0_48) * (input[i] > (192+offset3));

int rt3_0_49 = (input[i] > (197+offset3));
int t3_0_49 = rt3_0_49 * (2 + (input[i] > (198+offset3)));
t3_0_49 += (1 - rt3_0_49) * (input[i] > (196+offset3));

int rt3_0_50 = (input[i] > (201+offset3));
int t3_0_50 = rt3_0_50 * (2 + (input[i] > (202+offset3)));
t3_0_50 += (1 - rt3_0_50) * (input[i] > (200+offset3));

int rt3_0_51 = (input[i] > (205+offset3));
int t3_0_51 = rt3_0_51 * (2 + (input[i] > (206+offset3)));
t3_0_51 += (1 - rt3_0_51) * (input[i] > (204+offset3));

int rt3_0_52 = (input[i] > (209+offset3));
int t3_0_52 = rt3_0_52 * (2 + (input[i] > (210+offset3)));
t3_0_52 += (1 - rt3_0_52) * (input[i] > (208+offset3));

int rt3_0_53 = (input[i] > (213+offset3));
int t3_0_53 = rt3_0_53 * (2 + (input[i] > (214+offset3)));
t3_0_53 += (1 - rt3_0_53) * (input[i] > (212+offset3));

int rt3_0_54 = (input[i] > (217+offset3));
int t3_0_54 = rt3_0_54 * (2 + (input[i] > (218+offset3)));
t3_0_54 += (1 - rt3_0_54) * (input[i] > (216+offset3));

int rt3_0_55 = (input[i] > (221+offset3));
int t3_0_55 = rt3_0_55 * (2 + (input[i] > (222+offset3)));
t3_0_55 += (1 - rt3_0_55) * (input[i] > (220+offset3));

int rt3_0_56 = (input[i] > (225+offset3));
int t3_0_56 = rt3_0_56 * (2 + (input[i] > (226+offset3)));
t3_0_56 += (1 - rt3_0_56) * (input[i] > (224+offset3));

int rt3_0_57 = (input[i] > (229+offset3));
int t3_0_57 = rt3_0_57 * (2 + (input[i] > (230+offset3)));
t3_0_57 += (1 - rt3_0_57) * (input[i] > (228+offset3));

int rt3_0_58 = (input[i] > (233+offset3));
int t3_0_58 = rt3_0_58 * (2 + (input[i] > (234+offset3)));
t3_0_58 += (1 - rt3_0_58) * (input[i] > (232+offset3));

int rt3_0_59 = (input[i] > (237+offset3));
int t3_0_59 = rt3_0_59 * (2 + (input[i] > (238+offset3)));
t3_0_59 += (1 - rt3_0_59) * (input[i] > (236+offset3));

int rt3_0_60 = (input[i] > (241+offset3));
int t3_0_60 = rt3_0_60 * (2 + (input[i] > (242+offset3)));
t3_0_60 += (1 - rt3_0_60) * (input[i] > (240+offset3));

int rt3_0_61 = (input[i] > (245+offset3));
int t3_0_61 = rt3_0_61 * (2 + (input[i] > (246+offset3)));
t3_0_61 += (1 - rt3_0_61) * (input[i] > (244+offset3));

int rt3_0_62 = (input[i] > (249+offset3));
int t3_0_62 = rt3_0_62 * (2 + (input[i] > (250+offset3)));
t3_0_62 += (1 - rt3_0_62) * (input[i] > (248+offset3));

int rt3_0_63 = (input[i] > (253+offset3));
int t3_0_63 = rt3_0_63 * (2 + (input[i] > (254+offset3)));
t3_0_63 += (1 - rt3_0_63) * (input[i] > (252+offset3));

int rt3_0_64 = (input[i] > (257+offset3));
int t3_0_64 = rt3_0_64 * (2 + (input[i] > (258+offset3)));
t3_0_64 += (1 - rt3_0_64) * (input[i] > (256+offset3));

int rt3_0_65 = (input[i] > (261+offset3));
int t3_0_65 = rt3_0_65 * (2 + (input[i] > (262+offset3)));
t3_0_65 += (1 - rt3_0_65) * (input[i] > (260+offset3));

int rt3_0_66 = (input[i] > (265+offset3));
int t3_0_66 = rt3_0_66 * (2 + (input[i] > (266+offset3)));
t3_0_66 += (1 - rt3_0_66) * (input[i] > (264+offset3));

int rt3_0_67 = (input[i] > (269+offset3));
int t3_0_67 = rt3_0_67 * (2 + (input[i] > (270+offset3)));
t3_0_67 += (1 - rt3_0_67) * (input[i] > (268+offset3));

int rt3_0_68 = (input[i] > (273+offset3));
int t3_0_68 = rt3_0_68 * (2 + (input[i] > (274+offset3)));
t3_0_68 += (1 - rt3_0_68) * (input[i] > (272+offset3));

int rt3_0_69 = (input[i] > (277+offset3));
int t3_0_69 = rt3_0_69 * (2 + (input[i] > (278+offset3)));
t3_0_69 += (1 - rt3_0_69) * (input[i] > (276+offset3));

int rt3_0_70 = (input[i] > (281+offset3));
int t3_0_70 = rt3_0_70 * (2 + (input[i] > (282+offset3)));
t3_0_70 += (1 - rt3_0_70) * (input[i] > (280+offset3));

int rt3_0_71 = (input[i] > (285+offset3));
int t3_0_71 = rt3_0_71 * (2 + (input[i] > (286+offset3)));
t3_0_71 += (1 - rt3_0_71) * (input[i] > (284+offset3));

int rt3_0_72 = (input[i] > (289+offset3));
int t3_0_72 = rt3_0_72 * (2 + (input[i] > (290+offset3)));
t3_0_72 += (1 - rt3_0_72) * (input[i] > (288+offset3));

int rt3_0_73 = (input[i] > (293+offset3));
int t3_0_73 = rt3_0_73 * (2 + (input[i] > (294+offset3)));
t3_0_73 += (1 - rt3_0_73) * (input[i] > (292+offset3));

int rt3_0_74 = (input[i] > (297+offset3));
int t3_0_74 = rt3_0_74 * (2 + (input[i] > (298+offset3)));
t3_0_74 += (1 - rt3_0_74) * (input[i] > (296+offset3));

int rt3_0_75 = (input[i] > (301+offset3));
int t3_0_75 = rt3_0_75 * (2 + (input[i] > (302+offset3)));
t3_0_75 += (1 - rt3_0_75) * (input[i] > (300+offset3));

int rt3_0_76 = (input[i] > (305+offset3));
int t3_0_76 = rt3_0_76 * (2 + (input[i] > (306+offset3)));
t3_0_76 += (1 - rt3_0_76) * (input[i] > (304+offset3));

int rt3_0_77 = (input[i] > (309+offset3));
int t3_0_77 = rt3_0_77 * (2 + (input[i] > (310+offset3)));
t3_0_77 += (1 - rt3_0_77) * (input[i] > (308+offset3));

int rt3_0_78 = (input[i] > (313+offset3));
int t3_0_78 = rt3_0_78 * (2 + (input[i] > (314+offset3)));
t3_0_78 += (1 - rt3_0_78) * (input[i] > (312+offset3));

int rt3_0_79 = (input[i] > (317+offset3));
int t3_0_79 = rt3_0_79 * (2 + (input[i] > (318+offset3)));
t3_0_79 += (1 - rt3_0_79) * (input[i] > (316+offset3));

int rt3_0_80 = (input[i] > (321+offset3));
int t3_0_80 = rt3_0_80 * (2 + (input[i] > (322+offset3)));
t3_0_80 += (1 - rt3_0_80) * (input[i] > (320+offset3));

int rt3_0_81 = (input[i] > (325+offset3));
int t3_0_81 = rt3_0_81 * (2 + (input[i] > (326+offset3)));
t3_0_81 += (1 - rt3_0_81) * (input[i] > (324+offset3));

int rt3_0_82 = (input[i] > (329+offset3));
int t3_0_82 = rt3_0_82 * (2 + (input[i] > (330+offset3)));
t3_0_82 += (1 - rt3_0_82) * (input[i] > (328+offset3));

int rt3_0_83 = (input[i] > (333+offset3));
int t3_0_83 = rt3_0_83 * (2 + (input[i] > (334+offset3)));
t3_0_83 += (1 - rt3_0_83) * (input[i] > (332+offset3));

int rt3_0_84 = (input[i] > (337+offset3));
int t3_0_84 = rt3_0_84 * (2 + (input[i] > (338+offset3)));
t3_0_84 += (1 - rt3_0_84) * (input[i] > (336+offset3));

int rt3_0_85 = (input[i] > (341+offset3));
int t3_0_85 = rt3_0_85 * (2 + (input[i] > (342+offset3)));
t3_0_85 += (1 - rt3_0_85) * (input[i] > (340+offset3));

int rt3_0_86 = (input[i] > (345+offset3));
int t3_0_86 = rt3_0_86 * (2 + (input[i] > (346+offset3)));
t3_0_86 += (1 - rt3_0_86) * (input[i] > (344+offset3));

int rt3_0_87 = (input[i] > (349+offset3));
int t3_0_87 = rt3_0_87 * (2 + (input[i] > (350+offset3)));
t3_0_87 += (1 - rt3_0_87) * (input[i] > (348+offset3));

int rt3_0_88 = (input[i] > (353+offset3));
int t3_0_88 = rt3_0_88 * (2 + (input[i] > (354+offset3)));
t3_0_88 += (1 - rt3_0_88) * (input[i] > (352+offset3));

int rt3_0_89 = (input[i] > (357+offset3));
int t3_0_89 = rt3_0_89 * (2 + (input[i] > (358+offset3)));
t3_0_89 += (1 - rt3_0_89) * (input[i] > (356+offset3));

int rt3_0_90 = (input[i] > (361+offset3));
int t3_0_90 = rt3_0_90 * (2 + (input[i] > (362+offset3)));
t3_0_90 += (1 - rt3_0_90) * (input[i] > (360+offset3));

int rt3_0_91 = (input[i] > (365+offset3));
int t3_0_91 = rt3_0_91 * (2 + (input[i] > (366+offset3)));
t3_0_91 += (1 - rt3_0_91) * (input[i] > (364+offset3));

int rt3_0_92 = (input[i] > (369+offset3));
int t3_0_92 = rt3_0_92 * (2 + (input[i] > (370+offset3)));
t3_0_92 += (1 - rt3_0_92) * (input[i] > (368+offset3));

int rt3_0_93 = (input[i] > (373+offset3));
int t3_0_93 = rt3_0_93 * (2 + (input[i] > (374+offset3)));
t3_0_93 += (1 - rt3_0_93) * (input[i] > (372+offset3));

int rt3_0_94 = (input[i] > (377+offset3));
int t3_0_94 = rt3_0_94 * (2 + (input[i] > (378+offset3)));
t3_0_94 += (1 - rt3_0_94) * (input[i] > (376+offset3));

int rt3_0_95 = (input[i] > (381+offset3));
int t3_0_95 = rt3_0_95 * (2 + (input[i] > (382+offset3)));
t3_0_95 += (1 - rt3_0_95) * (input[i] > (380+offset3));

int rt3_0_96 = (input[i] > (385+offset3));
int t3_0_96 = rt3_0_96 * (2 + (input[i] > (386+offset3)));
t3_0_96 += (1 - rt3_0_96) * (input[i] > (384+offset3));

int rt3_0_97 = (input[i] > (389+offset3));
int t3_0_97 = rt3_0_97 * (2 + (input[i] > (390+offset3)));
t3_0_97 += (1 - rt3_0_97) * (input[i] > (388+offset3));

int rt3_0_98 = (input[i] > (393+offset3));
int t3_0_98 = rt3_0_98 * (2 + (input[i] > (394+offset3)));
t3_0_98 += (1 - rt3_0_98) * (input[i] > (392+offset3));

int rt3_0_99 = (input[i] > (397+offset3));
int t3_0_99 = rt3_0_99 * (2 + (input[i] > (398+offset3)));
t3_0_99 += (1 - rt3_0_99) * (input[i] > (396+offset3));

int rt3_0_100 = (input[i] > (401+offset3));
int t3_0_100 = rt3_0_100 * (2 + (input[i] > (402+offset3)));
t3_0_100 += (1 - rt3_0_100) * (input[i] > (400+offset3));

int rt3_0_101 = (input[i] > (405+offset3));
int t3_0_101 = rt3_0_101 * (2 + (input[i] > (406+offset3)));
t3_0_101 += (1 - rt3_0_101) * (input[i] > (404+offset3));

int rt3_0_102 = (input[i] > (409+offset3));
int t3_0_102 = rt3_0_102 * (2 + (input[i] > (410+offset3)));
t3_0_102 += (1 - rt3_0_102) * (input[i] > (408+offset3));

int rt3_0_103 = (input[i] > (413+offset3));
int t3_0_103 = rt3_0_103 * (2 + (input[i] > (414+offset3)));
t3_0_103 += (1 - rt3_0_103) * (input[i] > (412+offset3));

int rt3_0_104 = (input[i] > (417+offset3));
int t3_0_104 = rt3_0_104 * (2 + (input[i] > (418+offset3)));
t3_0_104 += (1 - rt3_0_104) * (input[i] > (416+offset3));

int rt3_0_105 = (input[i] > (421+offset3));
int t3_0_105 = rt3_0_105 * (2 + (input[i] > (422+offset3)));
t3_0_105 += (1 - rt3_0_105) * (input[i] > (420+offset3));

int rt3_0_106 = (input[i] > (425+offset3));
int t3_0_106 = rt3_0_106 * (2 + (input[i] > (426+offset3)));
t3_0_106 += (1 - rt3_0_106) * (input[i] > (424+offset3));

int rt3_0_107 = (input[i] > (429+offset3));
int t3_0_107 = rt3_0_107 * (2 + (input[i] > (430+offset3)));
t3_0_107 += (1 - rt3_0_107) * (input[i] > (428+offset3));

int rt3_0_108 = (input[i] > (433+offset3));
int t3_0_108 = rt3_0_108 * (2 + (input[i] > (434+offset3)));
t3_0_108 += (1 - rt3_0_108) * (input[i] > (432+offset3));

int rt3_0_109 = (input[i] > (437+offset3));
int t3_0_109 = rt3_0_109 * (2 + (input[i] > (438+offset3)));
t3_0_109 += (1 - rt3_0_109) * (input[i] > (436+offset3));

int rt3_0_110 = (input[i] > (441+offset3));
int t3_0_110 = rt3_0_110 * (2 + (input[i] > (442+offset3)));
t3_0_110 += (1 - rt3_0_110) * (input[i] > (440+offset3));

int rt3_0_111 = (input[i] > (445+offset3));
int t3_0_111 = rt3_0_111 * (2 + (input[i] > (446+offset3)));
t3_0_111 += (1 - rt3_0_111) * (input[i] > (444+offset3));

int rt3_0_112 = (input[i] > (449+offset3));
int t3_0_112 = rt3_0_112 * (2 + (input[i] > (450+offset3)));
t3_0_112 += (1 - rt3_0_112) * (input[i] > (448+offset3));

int rt3_0_113 = (input[i] > (453+offset3));
int t3_0_113 = rt3_0_113 * (2 + (input[i] > (454+offset3)));
t3_0_113 += (1 - rt3_0_113) * (input[i] > (452+offset3));

int rt3_0_114 = (input[i] > (457+offset3));
int t3_0_114 = rt3_0_114 * (2 + (input[i] > (458+offset3)));
t3_0_114 += (1 - rt3_0_114) * (input[i] > (456+offset3));

int rt3_0_115 = (input[i] > (461+offset3));
int t3_0_115 = rt3_0_115 * (2 + (input[i] > (462+offset3)));
t3_0_115 += (1 - rt3_0_115) * (input[i] > (460+offset3));

int rt3_0_116 = (input[i] > (465+offset3));
int t3_0_116 = rt3_0_116 * (2 + (input[i] > (466+offset3)));
t3_0_116 += (1 - rt3_0_116) * (input[i] > (464+offset3));

int rt3_0_117 = (input[i] > (469+offset3));
int t3_0_117 = rt3_0_117 * (2 + (input[i] > (470+offset3)));
t3_0_117 += (1 - rt3_0_117) * (input[i] > (468+offset3));

int rt3_0_118 = (input[i] > (473+offset3));
int t3_0_118 = rt3_0_118 * (2 + (input[i] > (474+offset3)));
t3_0_118 += (1 - rt3_0_118) * (input[i] > (472+offset3));

int rt3_0_119 = (input[i] > (477+offset3));
int t3_0_119 = rt3_0_119 * (2 + (input[i] > (478+offset3)));
t3_0_119 += (1 - rt3_0_119) * (input[i] > (476+offset3));

int rt3_0_120 = (input[i] > (481+offset3));
int t3_0_120 = rt3_0_120 * (2 + (input[i] > (482+offset3)));
t3_0_120 += (1 - rt3_0_120) * (input[i] > (480+offset3));

int rt3_0_121 = (input[i] > (485+offset3));
int t3_0_121 = rt3_0_121 * (2 + (input[i] > (486+offset3)));
t3_0_121 += (1 - rt3_0_121) * (input[i] > (484+offset3));

int rt3_0_122 = (input[i] > (489+offset3));
int t3_0_122 = rt3_0_122 * (2 + (input[i] > (490+offset3)));
t3_0_122 += (1 - rt3_0_122) * (input[i] > (488+offset3));

int rt3_0_123 = (input[i] > (493+offset3));
int t3_0_123 = rt3_0_123 * (2 + (input[i] > (494+offset3)));
t3_0_123 += (1 - rt3_0_123) * (input[i] > (492+offset3));

int rt3_0_124 = (input[i] > (497+offset3));
int t3_0_124 = rt3_0_124 * (2 + (input[i] > (498+offset3)));
t3_0_124 += (1 - rt3_0_124) * (input[i] > (496+offset3));

int rt3_0_125 = (input[i] > (501+offset3));
int t3_0_125 = rt3_0_125 * (2 + (input[i] > (502+offset3)));
t3_0_125 += (1 - rt3_0_125) * (input[i] > (500+offset3));

int rt3_0_126 = (input[i] > (505+offset3));
int t3_0_126 = rt3_0_126 * (2 + (input[i] > (506+offset3)));
t3_0_126 += (1 - rt3_0_126) * (input[i] > (504+offset3));

int rt3_0_127 = (input[i] > (509+offset3));
int t3_0_127 = rt3_0_127 * (2 + (input[i] > (510+offset3)));
t3_0_127 += (1 - rt3_0_127) * (input[i] > (508+offset3));

int rt3_0_128 = (input[i] > (513+offset3));
int t3_0_128 = rt3_0_128 * (2 + (input[i] > (514+offset3)));
t3_0_128 += (1 - rt3_0_128) * (input[i] > (512+offset3));

int rt3_0_129 = (input[i] > (517+offset3));
int t3_0_129 = rt3_0_129 * (2 + (input[i] > (518+offset3)));
t3_0_129 += (1 - rt3_0_129) * (input[i] > (516+offset3));

int rt3_0_130 = (input[i] > (521+offset3));
int t3_0_130 = rt3_0_130 * (2 + (input[i] > (522+offset3)));
t3_0_130 += (1 - rt3_0_130) * (input[i] > (520+offset3));

int rt3_0_131 = (input[i] > (525+offset3));
int t3_0_131 = rt3_0_131 * (2 + (input[i] > (526+offset3)));
t3_0_131 += (1 - rt3_0_131) * (input[i] > (524+offset3));

int rt3_0_132 = (input[i] > (529+offset3));
int t3_0_132 = rt3_0_132 * (2 + (input[i] > (530+offset3)));
t3_0_132 += (1 - rt3_0_132) * (input[i] > (528+offset3));

int rt3_0_133 = (input[i] > (533+offset3));
int t3_0_133 = rt3_0_133 * (2 + (input[i] > (534+offset3)));
t3_0_133 += (1 - rt3_0_133) * (input[i] > (532+offset3));

int rt3_0_134 = (input[i] > (537+offset3));
int t3_0_134 = rt3_0_134 * (2 + (input[i] > (538+offset3)));
t3_0_134 += (1 - rt3_0_134) * (input[i] > (536+offset3));

int rt3_0_135 = (input[i] > (541+offset3));
int t3_0_135 = rt3_0_135 * (2 + (input[i] > (542+offset3)));
t3_0_135 += (1 - rt3_0_135) * (input[i] > (540+offset3));

int rt3_0_136 = (input[i] > (545+offset3));
int t3_0_136 = rt3_0_136 * (2 + (input[i] > (546+offset3)));
t3_0_136 += (1 - rt3_0_136) * (input[i] > (544+offset3));

int rt3_0_137 = (input[i] > (549+offset3));
int t3_0_137 = rt3_0_137 * (2 + (input[i] > (550+offset3)));
t3_0_137 += (1 - rt3_0_137) * (input[i] > (548+offset3));

int rt3_0_138 = (input[i] > (553+offset3));
int t3_0_138 = rt3_0_138 * (2 + (input[i] > (554+offset3)));
t3_0_138 += (1 - rt3_0_138) * (input[i] > (552+offset3));

int rt3_0_139 = (input[i] > (557+offset3));
int t3_0_139 = rt3_0_139 * (2 + (input[i] > (558+offset3)));
t3_0_139 += (1 - rt3_0_139) * (input[i] > (556+offset3));

int rt3_0_140 = (input[i] > (561+offset3));
int t3_0_140 = rt3_0_140 * (2 + (input[i] > (562+offset3)));
t3_0_140 += (1 - rt3_0_140) * (input[i] > (560+offset3));

int rt3_0_141 = (input[i] > (565+offset3));
int t3_0_141 = rt3_0_141 * (2 + (input[i] > (566+offset3)));
t3_0_141 += (1 - rt3_0_141) * (input[i] > (564+offset3));

int rt3_0_142 = (input[i] > (569+offset3));
int t3_0_142 = rt3_0_142 * (2 + (input[i] > (570+offset3)));
t3_0_142 += (1 - rt3_0_142) * (input[i] > (568+offset3));

int rt3_0_143 = (input[i] > (573+offset3));
int t3_0_143 = rt3_0_143 * (2 + (input[i] > (574+offset3)));
t3_0_143 += (1 - rt3_0_143) * (input[i] > (572+offset3));

int rt3_0_144 = (input[i] > (577+offset3));
int t3_0_144 = rt3_0_144 * (2 + (input[i] > (578+offset3)));
t3_0_144 += (1 - rt3_0_144) * (input[i] > (576+offset3));

int rt3_0_145 = (input[i] > (581+offset3));
int t3_0_145 = rt3_0_145 * (2 + (input[i] > (582+offset3)));
t3_0_145 += (1 - rt3_0_145) * (input[i] > (580+offset3));

int rt3_0_146 = (input[i] > (585+offset3));
int t3_0_146 = rt3_0_146 * (2 + (input[i] > (586+offset3)));
t3_0_146 += (1 - rt3_0_146) * (input[i] > (584+offset3));

int rt3_0_147 = (input[i] > (589+offset3));
int t3_0_147 = rt3_0_147 * (2 + (input[i] > (590+offset3)));
t3_0_147 += (1 - rt3_0_147) * (input[i] > (588+offset3));

int rt3_0_148 = (input[i] > (593+offset3));
int t3_0_148 = rt3_0_148 * (2 + (input[i] > (594+offset3)));
t3_0_148 += (1 - rt3_0_148) * (input[i] > (592+offset3));

int rt3_0_149 = (input[i] > (597+offset3));
int t3_0_149 = rt3_0_149 * (2 + (input[i] > (598+offset3)));
t3_0_149 += (1 - rt3_0_149) * (input[i] > (596+offset3));

int rt3_0_150 = (input[i] > (601+offset3));
int t3_0_150 = rt3_0_150 * (2 + (input[i] > (602+offset3)));
t3_0_150 += (1 - rt3_0_150) * (input[i] > (600+offset3));

int rt3_0_151 = (input[i] > (605+offset3));
int t3_0_151 = rt3_0_151 * (2 + (input[i] > (606+offset3)));
t3_0_151 += (1 - rt3_0_151) * (input[i] > (604+offset3));

int rt3_0_152 = (input[i] > (609+offset3));
int t3_0_152 = rt3_0_152 * (2 + (input[i] > (610+offset3)));
t3_0_152 += (1 - rt3_0_152) * (input[i] > (608+offset3));

int rt3_0_153 = (input[i] > (613+offset3));
int t3_0_153 = rt3_0_153 * (2 + (input[i] > (614+offset3)));
t3_0_153 += (1 - rt3_0_153) * (input[i] > (612+offset3));

int rt3_0_154 = (input[i] > (617+offset3));
int t3_0_154 = rt3_0_154 * (2 + (input[i] > (618+offset3)));
t3_0_154 += (1 - rt3_0_154) * (input[i] > (616+offset3));

int rt3_0_155 = (input[i] > (621+offset3));
int t3_0_155 = rt3_0_155 * (2 + (input[i] > (622+offset3)));
t3_0_155 += (1 - rt3_0_155) * (input[i] > (620+offset3));

int rt3_0_156 = (input[i] > (625+offset3));
int t3_0_156 = rt3_0_156 * (2 + (input[i] > (626+offset3)));
t3_0_156 += (1 - rt3_0_156) * (input[i] > (624+offset3));

int rt3_0_157 = (input[i] > (629+offset3));
int t3_0_157 = rt3_0_157 * (2 + (input[i] > (630+offset3)));
t3_0_157 += (1 - rt3_0_157) * (input[i] > (628+offset3));

int rt3_0_158 = (input[i] > (633+offset3));
int t3_0_158 = rt3_0_158 * (2 + (input[i] > (634+offset3)));
t3_0_158 += (1 - rt3_0_158) * (input[i] > (632+offset3));

int rt3_0_159 = (input[i] > (637+offset3));
int t3_0_159 = rt3_0_159 * (2 + (input[i] > (638+offset3)));
t3_0_159 += (1 - rt3_0_159) * (input[i] > (636+offset3));

int rt3_0_160 = (input[i] > (641+offset3));
int t3_0_160 = rt3_0_160 * (2 + (input[i] > (642+offset3)));
t3_0_160 += (1 - rt3_0_160) * (input[i] > (640+offset3));

int rt3_0_161 = (input[i] > (645+offset3));
int t3_0_161 = rt3_0_161 * (2 + (input[i] > (646+offset3)));
t3_0_161 += (1 - rt3_0_161) * (input[i] > (644+offset3));

int rt3_0_162 = (input[i] > (649+offset3));
int t3_0_162 = rt3_0_162 * (2 + (input[i] > (650+offset3)));
t3_0_162 += (1 - rt3_0_162) * (input[i] > (648+offset3));

int rt3_0_163 = (input[i] > (653+offset3));
int t3_0_163 = rt3_0_163 * (2 + (input[i] > (654+offset3)));
t3_0_163 += (1 - rt3_0_163) * (input[i] > (652+offset3));

int rt3_0_164 = (input[i] > (657+offset3));
int t3_0_164 = rt3_0_164 * (2 + (input[i] > (658+offset3)));
t3_0_164 += (1 - rt3_0_164) * (input[i] > (656+offset3));

int rt3_0_165 = (input[i] > (661+offset3));
int t3_0_165 = rt3_0_165 * (2 + (input[i] > (662+offset3)));
t3_0_165 += (1 - rt3_0_165) * (input[i] > (660+offset3));

int rt3_0_166 = (input[i] > (665+offset3));
int t3_0_166 = rt3_0_166 * (2 + (input[i] > (666+offset3)));
t3_0_166 += (1 - rt3_0_166) * (input[i] > (664+offset3));

int rt3_0_167 = (input[i] > (669+offset3));
int t3_0_167 = rt3_0_167 * (2 + (input[i] > (670+offset3)));
t3_0_167 += (1 - rt3_0_167) * (input[i] > (668+offset3));

int rt3_0_168 = (input[i] > (673+offset3));
int t3_0_168 = rt3_0_168 * (2 + (input[i] > (674+offset3)));
t3_0_168 += (1 - rt3_0_168) * (input[i] > (672+offset3));

int rt3_0_169 = (input[i] > (677+offset3));
int t3_0_169 = rt3_0_169 * (2 + (input[i] > (678+offset3)));
t3_0_169 += (1 - rt3_0_169) * (input[i] > (676+offset3));

int rt3_0_170 = (input[i] > (681+offset3));
int t3_0_170 = rt3_0_170 * (2 + (input[i] > (682+offset3)));
t3_0_170 += (1 - rt3_0_170) * (input[i] > (680+offset3));

int rt3_0_171 = (input[i] > (685+offset3));
int t3_0_171 = rt3_0_171 * (2 + (input[i] > (686+offset3)));
t3_0_171 += (1 - rt3_0_171) * (input[i] > (684+offset3));

int rt3_0_172 = (input[i] > (689+offset3));
int t3_0_172 = rt3_0_172 * (2 + (input[i] > (690+offset3)));
t3_0_172 += (1 - rt3_0_172) * (input[i] > (688+offset3));

int rt3_0_173 = (input[i] > (693+offset3));
int t3_0_173 = rt3_0_173 * (2 + (input[i] > (694+offset3)));
t3_0_173 += (1 - rt3_0_173) * (input[i] > (692+offset3));

int rt3_0_174 = (input[i] > (697+offset3));
int t3_0_174 = rt3_0_174 * (2 + (input[i] > (698+offset3)));
t3_0_174 += (1 - rt3_0_174) * (input[i] > (696+offset3));

int rt3_0_175 = (input[i] > (701+offset3));
int t3_0_175 = rt3_0_175 * (2 + (input[i] > (702+offset3)));
t3_0_175 += (1 - rt3_0_175) * (input[i] > (700+offset3));

int rt3_0_176 = (input[i] > (705+offset3));
int t3_0_176 = rt3_0_176 * (2 + (input[i] > (706+offset3)));
t3_0_176 += (1 - rt3_0_176) * (input[i] > (704+offset3));

int rt3_0_177 = (input[i] > (709+offset3));
int t3_0_177 = rt3_0_177 * (2 + (input[i] > (710+offset3)));
t3_0_177 += (1 - rt3_0_177) * (input[i] > (708+offset3));

int rt3_0_178 = (input[i] > (713+offset3));
int t3_0_178 = rt3_0_178 * (2 + (input[i] > (714+offset3)));
t3_0_178 += (1 - rt3_0_178) * (input[i] > (712+offset3));

int rt3_0_179 = (input[i] > (717+offset3));
int t3_0_179 = rt3_0_179 * (2 + (input[i] > (718+offset3)));
t3_0_179 += (1 - rt3_0_179) * (input[i] > (716+offset3));

int rt3_0_180 = (input[i] > (721+offset3));
int t3_0_180 = rt3_0_180 * (2 + (input[i] > (722+offset3)));
t3_0_180 += (1 - rt3_0_180) * (input[i] > (720+offset3));

int rt3_0_181 = (input[i] > (725+offset3));
int t3_0_181 = rt3_0_181 * (2 + (input[i] > (726+offset3)));
t3_0_181 += (1 - rt3_0_181) * (input[i] > (724+offset3));

int rt3_0_182 = (input[i] > (729+offset3));
int t3_0_182 = rt3_0_182 * (2 + (input[i] > (730+offset3)));
t3_0_182 += (1 - rt3_0_182) * (input[i] > (728+offset3));

int rt3_0_183 = (input[i] > (733+offset3));
int t3_0_183 = rt3_0_183 * (2 + (input[i] > (734+offset3)));
t3_0_183 += (1 - rt3_0_183) * (input[i] > (732+offset3));

int rt3_0_184 = (input[i] > (737+offset3));
int t3_0_184 = rt3_0_184 * (2 + (input[i] > (738+offset3)));
t3_0_184 += (1 - rt3_0_184) * (input[i] > (736+offset3));

int rt3_0_185 = (input[i] > (741+offset3));
int t3_0_185 = rt3_0_185 * (2 + (input[i] > (742+offset3)));
t3_0_185 += (1 - rt3_0_185) * (input[i] > (740+offset3));

int rt3_0_186 = (input[i] > (745+offset3));
int t3_0_186 = rt3_0_186 * (2 + (input[i] > (746+offset3)));
t3_0_186 += (1 - rt3_0_186) * (input[i] > (744+offset3));

int rt3_0_187 = (input[i] > (749+offset3));
int t3_0_187 = rt3_0_187 * (2 + (input[i] > (750+offset3)));
t3_0_187 += (1 - rt3_0_187) * (input[i] > (748+offset3));

int rt3_0_188 = (input[i] > (753+offset3));
int t3_0_188 = rt3_0_188 * (2 + (input[i] > (754+offset3)));
t3_0_188 += (1 - rt3_0_188) * (input[i] > (752+offset3));

int rt3_0_189 = (input[i] > (757+offset3));
int t3_0_189 = rt3_0_189 * (2 + (input[i] > (758+offset3)));
t3_0_189 += (1 - rt3_0_189) * (input[i] > (756+offset3));

int rt3_0_190 = (input[i] > (761+offset3));
int t3_0_190 = rt3_0_190 * (2 + (input[i] > (762+offset3)));
t3_0_190 += (1 - rt3_0_190) * (input[i] > (760+offset3));

int rt3_0_191 = (input[i] > (765+offset3));
int t3_0_191 = rt3_0_191 * (2 + (input[i] > (766+offset3)));
t3_0_191 += (1 - rt3_0_191) * (input[i] > (764+offset3));

int rt3_0_192 = (input[i] > (769+offset3));
int t3_0_192 = rt3_0_192 * (2 + (input[i] > (770+offset3)));
t3_0_192 += (1 - rt3_0_192) * (input[i] > (768+offset3));

int rt3_0_193 = (input[i] > (773+offset3));
int t3_0_193 = rt3_0_193 * (2 + (input[i] > (774+offset3)));
t3_0_193 += (1 - rt3_0_193) * (input[i] > (772+offset3));

int rt3_0_194 = (input[i] > (777+offset3));
int t3_0_194 = rt3_0_194 * (2 + (input[i] > (778+offset3)));
t3_0_194 += (1 - rt3_0_194) * (input[i] > (776+offset3));

int rt3_0_195 = (input[i] > (781+offset3));
int t3_0_195 = rt3_0_195 * (2 + (input[i] > (782+offset3)));
t3_0_195 += (1 - rt3_0_195) * (input[i] > (780+offset3));

int rt3_0_196 = (input[i] > (785+offset3));
int t3_0_196 = rt3_0_196 * (2 + (input[i] > (786+offset3)));
t3_0_196 += (1 - rt3_0_196) * (input[i] > (784+offset3));

int rt3_0_197 = (input[i] > (789+offset3));
int t3_0_197 = rt3_0_197 * (2 + (input[i] > (790+offset3)));
t3_0_197 += (1 - rt3_0_197) * (input[i] > (788+offset3));

int rt3_0_198 = (input[i] > (793+offset3));
int t3_0_198 = rt3_0_198 * (2 + (input[i] > (794+offset3)));
t3_0_198 += (1 - rt3_0_198) * (input[i] > (792+offset3));

int rt3_0_199 = (input[i] > (797+offset3));
int t3_0_199 = rt3_0_199 * (2 + (input[i] > (798+offset3)));
t3_0_199 += (1 - rt3_0_199) * (input[i] > (796+offset3));

int rt3_0_200 = (input[i] > (801+offset3));
int t3_0_200 = rt3_0_200 * (2 + (input[i] > (802+offset3)));
t3_0_200 += (1 - rt3_0_200) * (input[i] > (800+offset3));

int rt3_0_201 = (input[i] > (805+offset3));
int t3_0_201 = rt3_0_201 * (2 + (input[i] > (806+offset3)));
t3_0_201 += (1 - rt3_0_201) * (input[i] > (804+offset3));

int rt3_0_202 = (input[i] > (809+offset3));
int t3_0_202 = rt3_0_202 * (2 + (input[i] > (810+offset3)));
t3_0_202 += (1 - rt3_0_202) * (input[i] > (808+offset3));

int rt3_0_203 = (input[i] > (813+offset3));
int t3_0_203 = rt3_0_203 * (2 + (input[i] > (814+offset3)));
t3_0_203 += (1 - rt3_0_203) * (input[i] > (812+offset3));

int rt3_0_204 = (input[i] > (817+offset3));
int t3_0_204 = rt3_0_204 * (2 + (input[i] > (818+offset3)));
t3_0_204 += (1 - rt3_0_204) * (input[i] > (816+offset3));

int rt3_0_205 = (input[i] > (821+offset3));
int t3_0_205 = rt3_0_205 * (2 + (input[i] > (822+offset3)));
t3_0_205 += (1 - rt3_0_205) * (input[i] > (820+offset3));

int rt3_0_206 = (input[i] > (825+offset3));
int t3_0_206 = rt3_0_206 * (2 + (input[i] > (826+offset3)));
t3_0_206 += (1 - rt3_0_206) * (input[i] > (824+offset3));

int rt3_0_207 = (input[i] > (829+offset3));
int t3_0_207 = rt3_0_207 * (2 + (input[i] > (830+offset3)));
t3_0_207 += (1 - rt3_0_207) * (input[i] > (828+offset3));

int rt3_0_208 = (input[i] > (833+offset3));
int t3_0_208 = rt3_0_208 * (2 + (input[i] > (834+offset3)));
t3_0_208 += (1 - rt3_0_208) * (input[i] > (832+offset3));

int rt3_0_209 = (input[i] > (837+offset3));
int t3_0_209 = rt3_0_209 * (2 + (input[i] > (838+offset3)));
t3_0_209 += (1 - rt3_0_209) * (input[i] > (836+offset3));

int rt3_0_210 = (input[i] > (841+offset3));
int t3_0_210 = rt3_0_210 * (2 + (input[i] > (842+offset3)));
t3_0_210 += (1 - rt3_0_210) * (input[i] > (840+offset3));

int rt3_0_211 = (input[i] > (845+offset3));
int t3_0_211 = rt3_0_211 * (2 + (input[i] > (846+offset3)));
t3_0_211 += (1 - rt3_0_211) * (input[i] > (844+offset3));

int rt3_0_212 = (input[i] > (849+offset3));
int t3_0_212 = rt3_0_212 * (2 + (input[i] > (850+offset3)));
t3_0_212 += (1 - rt3_0_212) * (input[i] > (848+offset3));

int rt3_0_213 = (input[i] > (853+offset3));
int t3_0_213 = rt3_0_213 * (2 + (input[i] > (854+offset3)));
t3_0_213 += (1 - rt3_0_213) * (input[i] > (852+offset3));

int rt3_0_214 = (input[i] > (857+offset3));
int t3_0_214 = rt3_0_214 * (2 + (input[i] > (858+offset3)));
t3_0_214 += (1 - rt3_0_214) * (input[i] > (856+offset3));

int rt3_0_215 = (input[i] > (861+offset3));
int t3_0_215 = rt3_0_215 * (2 + (input[i] > (862+offset3)));
t3_0_215 += (1 - rt3_0_215) * (input[i] > (860+offset3));

int rt3_0_216 = (input[i] > (865+offset3));
int t3_0_216 = rt3_0_216 * (2 + (input[i] > (866+offset3)));
t3_0_216 += (1 - rt3_0_216) * (input[i] > (864+offset3));

int rt3_0_217 = (input[i] > (869+offset3));
int t3_0_217 = rt3_0_217 * (2 + (input[i] > (870+offset3)));
t3_0_217 += (1 - rt3_0_217) * (input[i] > (868+offset3));

int rt3_0_218 = (input[i] > (873+offset3));
int t3_0_218 = rt3_0_218 * (2 + (input[i] > (874+offset3)));
t3_0_218 += (1 - rt3_0_218) * (input[i] > (872+offset3));

int rt3_0_219 = (input[i] > (877+offset3));
int t3_0_219 = rt3_0_219 * (2 + (input[i] > (878+offset3)));
t3_0_219 += (1 - rt3_0_219) * (input[i] > (876+offset3));

int rt3_0_220 = (input[i] > (881+offset3));
int t3_0_220 = rt3_0_220 * (2 + (input[i] > (882+offset3)));
t3_0_220 += (1 - rt3_0_220) * (input[i] > (880+offset3));

int rt3_0_221 = (input[i] > (885+offset3));
int t3_0_221 = rt3_0_221 * (2 + (input[i] > (886+offset3)));
t3_0_221 += (1 - rt3_0_221) * (input[i] > (884+offset3));

int rt3_0_222 = (input[i] > (889+offset3));
int t3_0_222 = rt3_0_222 * (2 + (input[i] > (890+offset3)));
t3_0_222 += (1 - rt3_0_222) * (input[i] > (888+offset3));

int rt3_0_223 = (input[i] > (893+offset3));
int t3_0_223 = rt3_0_223 * (2 + (input[i] > (894+offset3)));
t3_0_223 += (1 - rt3_0_223) * (input[i] > (892+offset3));

int rt3_0_224 = (input[i] > (897+offset3));
int t3_0_224 = rt3_0_224 * (2 + (input[i] > (898+offset3)));
t3_0_224 += (1 - rt3_0_224) * (input[i] > (896+offset3));

int rt3_0_225 = (input[i] > (901+offset3));
int t3_0_225 = rt3_0_225 * (2 + (input[i] > (902+offset3)));
t3_0_225 += (1 - rt3_0_225) * (input[i] > (900+offset3));

int rt3_0_226 = (input[i] > (905+offset3));
int t3_0_226 = rt3_0_226 * (2 + (input[i] > (906+offset3)));
t3_0_226 += (1 - rt3_0_226) * (input[i] > (904+offset3));

int rt3_0_227 = (input[i] > (909+offset3));
int t3_0_227 = rt3_0_227 * (2 + (input[i] > (910+offset3)));
t3_0_227 += (1 - rt3_0_227) * (input[i] > (908+offset3));

int rt3_0_228 = (input[i] > (913+offset3));
int t3_0_228 = rt3_0_228 * (2 + (input[i] > (914+offset3)));
t3_0_228 += (1 - rt3_0_228) * (input[i] > (912+offset3));

int rt3_0_229 = (input[i] > (917+offset3));
int t3_0_229 = rt3_0_229 * (2 + (input[i] > (918+offset3)));
t3_0_229 += (1 - rt3_0_229) * (input[i] > (916+offset3));

int rt3_0_230 = (input[i] > (921+offset3));
int t3_0_230 = rt3_0_230 * (2 + (input[i] > (922+offset3)));
t3_0_230 += (1 - rt3_0_230) * (input[i] > (920+offset3));

int rt3_0_231 = (input[i] > (925+offset3));
int t3_0_231 = rt3_0_231 * (2 + (input[i] > (926+offset3)));
t3_0_231 += (1 - rt3_0_231) * (input[i] > (924+offset3));

int rt3_0_232 = (input[i] > (929+offset3));
int t3_0_232 = rt3_0_232 * (2 + (input[i] > (930+offset3)));
t3_0_232 += (1 - rt3_0_232) * (input[i] > (928+offset3));

int rt3_0_233 = (input[i] > (933+offset3));
int t3_0_233 = rt3_0_233 * (2 + (input[i] > (934+offset3)));
t3_0_233 += (1 - rt3_0_233) * (input[i] > (932+offset3));

int rt3_0_234 = (input[i] > (937+offset3));
int t3_0_234 = rt3_0_234 * (2 + (input[i] > (938+offset3)));
t3_0_234 += (1 - rt3_0_234) * (input[i] > (936+offset3));

int rt3_0_235 = (input[i] > (941+offset3));
int t3_0_235 = rt3_0_235 * (2 + (input[i] > (942+offset3)));
t3_0_235 += (1 - rt3_0_235) * (input[i] > (940+offset3));

int rt3_0_236 = (input[i] > (945+offset3));
int t3_0_236 = rt3_0_236 * (2 + (input[i] > (946+offset3)));
t3_0_236 += (1 - rt3_0_236) * (input[i] > (944+offset3));

int rt3_0_237 = (input[i] > (949+offset3));
int t3_0_237 = rt3_0_237 * (2 + (input[i] > (950+offset3)));
t3_0_237 += (1 - rt3_0_237) * (input[i] > (948+offset3));

int rt3_0_238 = (input[i] > (953+offset3));
int t3_0_238 = rt3_0_238 * (2 + (input[i] > (954+offset3)));
t3_0_238 += (1 - rt3_0_238) * (input[i] > (952+offset3));

int rt3_0_239 = (input[i] > (957+offset3));
int t3_0_239 = rt3_0_239 * (2 + (input[i] > (958+offset3)));
t3_0_239 += (1 - rt3_0_239) * (input[i] > (956+offset3));

int rt3_0_240 = (input[i] > (961+offset3));
int t3_0_240 = rt3_0_240 * (2 + (input[i] > (962+offset3)));
t3_0_240 += (1 - rt3_0_240) * (input[i] > (960+offset3));

int rt3_0_241 = (input[i] > (965+offset3));
int t3_0_241 = rt3_0_241 * (2 + (input[i] > (966+offset3)));
t3_0_241 += (1 - rt3_0_241) * (input[i] > (964+offset3));

int rt3_0_242 = (input[i] > (969+offset3));
int t3_0_242 = rt3_0_242 * (2 + (input[i] > (970+offset3)));
t3_0_242 += (1 - rt3_0_242) * (input[i] > (968+offset3));

int rt3_0_243 = (input[i] > (973+offset3));
int t3_0_243 = rt3_0_243 * (2 + (input[i] > (974+offset3)));
t3_0_243 += (1 - rt3_0_243) * (input[i] > (972+offset3));

int rt3_0_244 = (input[i] > (977+offset3));
int t3_0_244 = rt3_0_244 * (2 + (input[i] > (978+offset3)));
t3_0_244 += (1 - rt3_0_244) * (input[i] > (976+offset3));

int rt3_0_245 = (input[i] > (981+offset3));
int t3_0_245 = rt3_0_245 * (2 + (input[i] > (982+offset3)));
t3_0_245 += (1 - rt3_0_245) * (input[i] > (980+offset3));

int rt3_0_246 = (input[i] > (985+offset3));
int t3_0_246 = rt3_0_246 * (2 + (input[i] > (986+offset3)));
t3_0_246 += (1 - rt3_0_246) * (input[i] > (984+offset3));

int rt3_0_247 = (input[i] > (989+offset3));
int t3_0_247 = rt3_0_247 * (2 + (input[i] > (990+offset3)));
t3_0_247 += (1 - rt3_0_247) * (input[i] > (988+offset3));

int rt3_0_248 = (input[i] > (993+offset3));
int t3_0_248 = rt3_0_248 * (2 + (input[i] > (994+offset3)));
t3_0_248 += (1 - rt3_0_248) * (input[i] > (992+offset3));

int rt3_0_249 = (input[i] > (997+offset3));
int t3_0_249 = rt3_0_249 * (2 + (input[i] > (998+offset3)));
t3_0_249 += (1 - rt3_0_249) * (input[i] > (996+offset3));

int rt3_0_250 = (input[i] > (1001+offset3));
int t3_0_250 = rt3_0_250 * (2 + (input[i] > (1002+offset3)));
t3_0_250 += (1 - rt3_0_250) * (input[i] > (1000+offset3));

int rt3_0_251 = (input[i] > (1005+offset3));
int t3_0_251 = rt3_0_251 * (2 + (input[i] > (1006+offset3)));
t3_0_251 += (1 - rt3_0_251) * (input[i] > (1004+offset3));

int rt3_0_252 = (input[i] > (1009+offset3));
int t3_0_252 = rt3_0_252 * (2 + (input[i] > (1010+offset3)));
t3_0_252 += (1 - rt3_0_252) * (input[i] > (1008+offset3));

int rt3_0_253 = (input[i] > (1013+offset3));
int t3_0_253 = rt3_0_253 * (2 + (input[i] > (1014+offset3)));
t3_0_253 += (1 - rt3_0_253) * (input[i] > (1012+offset3));

int rt3_0_254 = (input[i] > (1017+offset3));
int t3_0_254 = rt3_0_254 * (2 + (input[i] > (1018+offset3)));
t3_0_254 += (1 - rt3_0_254) * (input[i] > (1016+offset3));

int rt3_0_255 = (input[i] > (1021+offset3));
int t3_0_255 = rt3_0_255 * (2 + (input[i] > (1022+offset3)));
t3_0_255 += (1 - rt3_0_255) * (input[i] > (1020+offset3));

int rt3_1_0 = (input[i] > (3+offset3));
int t3_1_0 = rt3_1_0 * (4 + t3_0_1);
t3_1_0 += (1 - rt3_1_0) * t3_0_0;

int rt3_1_1 = (input[i] > (11+offset3));
int t3_1_1 = rt3_1_1 * (4 + t3_0_3);
t3_1_1 += (1 - rt3_1_1) * t3_0_2;

int rt3_1_2 = (input[i] > (19+offset3));
int t3_1_2 = rt3_1_2 * (4 + t3_0_5);
t3_1_2 += (1 - rt3_1_2) * t3_0_4;

int rt3_1_3 = (input[i] > (27+offset3));
int t3_1_3 = rt3_1_3 * (4 + t3_0_7);
t3_1_3 += (1 - rt3_1_3) * t3_0_6;

int rt3_1_4 = (input[i] > (35+offset3));
int t3_1_4 = rt3_1_4 * (4 + t3_0_9);
t3_1_4 += (1 - rt3_1_4) * t3_0_8;

int rt3_1_5 = (input[i] > (43+offset3));
int t3_1_5 = rt3_1_5 * (4 + t3_0_11);
t3_1_5 += (1 - rt3_1_5) * t3_0_10;

int rt3_1_6 = (input[i] > (51+offset3));
int t3_1_6 = rt3_1_6 * (4 + t3_0_13);
t3_1_6 += (1 - rt3_1_6) * t3_0_12;

int rt3_1_7 = (input[i] > (59+offset3));
int t3_1_7 = rt3_1_7 * (4 + t3_0_15);
t3_1_7 += (1 - rt3_1_7) * t3_0_14;

int rt3_1_8 = (input[i] > (67+offset3));
int t3_1_8 = rt3_1_8 * (4 + t3_0_17);
t3_1_8 += (1 - rt3_1_8) * t3_0_16;

int rt3_1_9 = (input[i] > (75+offset3));
int t3_1_9 = rt3_1_9 * (4 + t3_0_19);
t3_1_9 += (1 - rt3_1_9) * t3_0_18;

int rt3_1_10 = (input[i] > (83+offset3));
int t3_1_10 = rt3_1_10 * (4 + t3_0_21);
t3_1_10 += (1 - rt3_1_10) * t3_0_20;

int rt3_1_11 = (input[i] > (91+offset3));
int t3_1_11 = rt3_1_11 * (4 + t3_0_23);
t3_1_11 += (1 - rt3_1_11) * t3_0_22;

int rt3_1_12 = (input[i] > (99+offset3));
int t3_1_12 = rt3_1_12 * (4 + t3_0_25);
t3_1_12 += (1 - rt3_1_12) * t3_0_24;

int rt3_1_13 = (input[i] > (107+offset3));
int t3_1_13 = rt3_1_13 * (4 + t3_0_27);
t3_1_13 += (1 - rt3_1_13) * t3_0_26;

int rt3_1_14 = (input[i] > (115+offset3));
int t3_1_14 = rt3_1_14 * (4 + t3_0_29);
t3_1_14 += (1 - rt3_1_14) * t3_0_28;

int rt3_1_15 = (input[i] > (123+offset3));
int t3_1_15 = rt3_1_15 * (4 + t3_0_31);
t3_1_15 += (1 - rt3_1_15) * t3_0_30;

int rt3_1_16 = (input[i] > (131+offset3));
int t3_1_16 = rt3_1_16 * (4 + t3_0_33);
t3_1_16 += (1 - rt3_1_16) * t3_0_32;

int rt3_1_17 = (input[i] > (139+offset3));
int t3_1_17 = rt3_1_17 * (4 + t3_0_35);
t3_1_17 += (1 - rt3_1_17) * t3_0_34;

int rt3_1_18 = (input[i] > (147+offset3));
int t3_1_18 = rt3_1_18 * (4 + t3_0_37);
t3_1_18 += (1 - rt3_1_18) * t3_0_36;

int rt3_1_19 = (input[i] > (155+offset3));
int t3_1_19 = rt3_1_19 * (4 + t3_0_39);
t3_1_19 += (1 - rt3_1_19) * t3_0_38;

int rt3_1_20 = (input[i] > (163+offset3));
int t3_1_20 = rt3_1_20 * (4 + t3_0_41);
t3_1_20 += (1 - rt3_1_20) * t3_0_40;

int rt3_1_21 = (input[i] > (171+offset3));
int t3_1_21 = rt3_1_21 * (4 + t3_0_43);
t3_1_21 += (1 - rt3_1_21) * t3_0_42;

int rt3_1_22 = (input[i] > (179+offset3));
int t3_1_22 = rt3_1_22 * (4 + t3_0_45);
t3_1_22 += (1 - rt3_1_22) * t3_0_44;

int rt3_1_23 = (input[i] > (187+offset3));
int t3_1_23 = rt3_1_23 * (4 + t3_0_47);
t3_1_23 += (1 - rt3_1_23) * t3_0_46;

int rt3_1_24 = (input[i] > (195+offset3));
int t3_1_24 = rt3_1_24 * (4 + t3_0_49);
t3_1_24 += (1 - rt3_1_24) * t3_0_48;

int rt3_1_25 = (input[i] > (203+offset3));
int t3_1_25 = rt3_1_25 * (4 + t3_0_51);
t3_1_25 += (1 - rt3_1_25) * t3_0_50;

int rt3_1_26 = (input[i] > (211+offset3));
int t3_1_26 = rt3_1_26 * (4 + t3_0_53);
t3_1_26 += (1 - rt3_1_26) * t3_0_52;

int rt3_1_27 = (input[i] > (219+offset3));
int t3_1_27 = rt3_1_27 * (4 + t3_0_55);
t3_1_27 += (1 - rt3_1_27) * t3_0_54;

int rt3_1_28 = (input[i] > (227+offset3));
int t3_1_28 = rt3_1_28 * (4 + t3_0_57);
t3_1_28 += (1 - rt3_1_28) * t3_0_56;

int rt3_1_29 = (input[i] > (235+offset3));
int t3_1_29 = rt3_1_29 * (4 + t3_0_59);
t3_1_29 += (1 - rt3_1_29) * t3_0_58;

int rt3_1_30 = (input[i] > (243+offset3));
int t3_1_30 = rt3_1_30 * (4 + t3_0_61);
t3_1_30 += (1 - rt3_1_30) * t3_0_60;

int rt3_1_31 = (input[i] > (251+offset3));
int t3_1_31 = rt3_1_31 * (4 + t3_0_63);
t3_1_31 += (1 - rt3_1_31) * t3_0_62;

int rt3_1_32 = (input[i] > (259+offset3));
int t3_1_32 = rt3_1_32 * (4 + t3_0_65);
t3_1_32 += (1 - rt3_1_32) * t3_0_64;

int rt3_1_33 = (input[i] > (267+offset3));
int t3_1_33 = rt3_1_33 * (4 + t3_0_67);
t3_1_33 += (1 - rt3_1_33) * t3_0_66;

int rt3_1_34 = (input[i] > (275+offset3));
int t3_1_34 = rt3_1_34 * (4 + t3_0_69);
t3_1_34 += (1 - rt3_1_34) * t3_0_68;

int rt3_1_35 = (input[i] > (283+offset3));
int t3_1_35 = rt3_1_35 * (4 + t3_0_71);
t3_1_35 += (1 - rt3_1_35) * t3_0_70;

int rt3_1_36 = (input[i] > (291+offset3));
int t3_1_36 = rt3_1_36 * (4 + t3_0_73);
t3_1_36 += (1 - rt3_1_36) * t3_0_72;

int rt3_1_37 = (input[i] > (299+offset3));
int t3_1_37 = rt3_1_37 * (4 + t3_0_75);
t3_1_37 += (1 - rt3_1_37) * t3_0_74;

int rt3_1_38 = (input[i] > (307+offset3));
int t3_1_38 = rt3_1_38 * (4 + t3_0_77);
t3_1_38 += (1 - rt3_1_38) * t3_0_76;

int rt3_1_39 = (input[i] > (315+offset3));
int t3_1_39 = rt3_1_39 * (4 + t3_0_79);
t3_1_39 += (1 - rt3_1_39) * t3_0_78;

int rt3_1_40 = (input[i] > (323+offset3));
int t3_1_40 = rt3_1_40 * (4 + t3_0_81);
t3_1_40 += (1 - rt3_1_40) * t3_0_80;

int rt3_1_41 = (input[i] > (331+offset3));
int t3_1_41 = rt3_1_41 * (4 + t3_0_83);
t3_1_41 += (1 - rt3_1_41) * t3_0_82;

int rt3_1_42 = (input[i] > (339+offset3));
int t3_1_42 = rt3_1_42 * (4 + t3_0_85);
t3_1_42 += (1 - rt3_1_42) * t3_0_84;

int rt3_1_43 = (input[i] > (347+offset3));
int t3_1_43 = rt3_1_43 * (4 + t3_0_87);
t3_1_43 += (1 - rt3_1_43) * t3_0_86;

int rt3_1_44 = (input[i] > (355+offset3));
int t3_1_44 = rt3_1_44 * (4 + t3_0_89);
t3_1_44 += (1 - rt3_1_44) * t3_0_88;

int rt3_1_45 = (input[i] > (363+offset3));
int t3_1_45 = rt3_1_45 * (4 + t3_0_91);
t3_1_45 += (1 - rt3_1_45) * t3_0_90;

int rt3_1_46 = (input[i] > (371+offset3));
int t3_1_46 = rt3_1_46 * (4 + t3_0_93);
t3_1_46 += (1 - rt3_1_46) * t3_0_92;

int rt3_1_47 = (input[i] > (379+offset3));
int t3_1_47 = rt3_1_47 * (4 + t3_0_95);
t3_1_47 += (1 - rt3_1_47) * t3_0_94;

int rt3_1_48 = (input[i] > (387+offset3));
int t3_1_48 = rt3_1_48 * (4 + t3_0_97);
t3_1_48 += (1 - rt3_1_48) * t3_0_96;

int rt3_1_49 = (input[i] > (395+offset3));
int t3_1_49 = rt3_1_49 * (4 + t3_0_99);
t3_1_49 += (1 - rt3_1_49) * t3_0_98;

int rt3_1_50 = (input[i] > (403+offset3));
int t3_1_50 = rt3_1_50 * (4 + t3_0_101);
t3_1_50 += (1 - rt3_1_50) * t3_0_100;

int rt3_1_51 = (input[i] > (411+offset3));
int t3_1_51 = rt3_1_51 * (4 + t3_0_103);
t3_1_51 += (1 - rt3_1_51) * t3_0_102;

int rt3_1_52 = (input[i] > (419+offset3));
int t3_1_52 = rt3_1_52 * (4 + t3_0_105);
t3_1_52 += (1 - rt3_1_52) * t3_0_104;

int rt3_1_53 = (input[i] > (427+offset3));
int t3_1_53 = rt3_1_53 * (4 + t3_0_107);
t3_1_53 += (1 - rt3_1_53) * t3_0_106;

int rt3_1_54 = (input[i] > (435+offset3));
int t3_1_54 = rt3_1_54 * (4 + t3_0_109);
t3_1_54 += (1 - rt3_1_54) * t3_0_108;

int rt3_1_55 = (input[i] > (443+offset3));
int t3_1_55 = rt3_1_55 * (4 + t3_0_111);
t3_1_55 += (1 - rt3_1_55) * t3_0_110;

int rt3_1_56 = (input[i] > (451+offset3));
int t3_1_56 = rt3_1_56 * (4 + t3_0_113);
t3_1_56 += (1 - rt3_1_56) * t3_0_112;

int rt3_1_57 = (input[i] > (459+offset3));
int t3_1_57 = rt3_1_57 * (4 + t3_0_115);
t3_1_57 += (1 - rt3_1_57) * t3_0_114;

int rt3_1_58 = (input[i] > (467+offset3));
int t3_1_58 = rt3_1_58 * (4 + t3_0_117);
t3_1_58 += (1 - rt3_1_58) * t3_0_116;

int rt3_1_59 = (input[i] > (475+offset3));
int t3_1_59 = rt3_1_59 * (4 + t3_0_119);
t3_1_59 += (1 - rt3_1_59) * t3_0_118;

int rt3_1_60 = (input[i] > (483+offset3));
int t3_1_60 = rt3_1_60 * (4 + t3_0_121);
t3_1_60 += (1 - rt3_1_60) * t3_0_120;

int rt3_1_61 = (input[i] > (491+offset3));
int t3_1_61 = rt3_1_61 * (4 + t3_0_123);
t3_1_61 += (1 - rt3_1_61) * t3_0_122;

int rt3_1_62 = (input[i] > (499+offset3));
int t3_1_62 = rt3_1_62 * (4 + t3_0_125);
t3_1_62 += (1 - rt3_1_62) * t3_0_124;

int rt3_1_63 = (input[i] > (507+offset3));
int t3_1_63 = rt3_1_63 * (4 + t3_0_127);
t3_1_63 += (1 - rt3_1_63) * t3_0_126;

int rt3_1_64 = (input[i] > (515+offset3));
int t3_1_64 = rt3_1_64 * (4 + t3_0_129);
t3_1_64 += (1 - rt3_1_64) * t3_0_128;

int rt3_1_65 = (input[i] > (523+offset3));
int t3_1_65 = rt3_1_65 * (4 + t3_0_131);
t3_1_65 += (1 - rt3_1_65) * t3_0_130;

int rt3_1_66 = (input[i] > (531+offset3));
int t3_1_66 = rt3_1_66 * (4 + t3_0_133);
t3_1_66 += (1 - rt3_1_66) * t3_0_132;

int rt3_1_67 = (input[i] > (539+offset3));
int t3_1_67 = rt3_1_67 * (4 + t3_0_135);
t3_1_67 += (1 - rt3_1_67) * t3_0_134;

int rt3_1_68 = (input[i] > (547+offset3));
int t3_1_68 = rt3_1_68 * (4 + t3_0_137);
t3_1_68 += (1 - rt3_1_68) * t3_0_136;

int rt3_1_69 = (input[i] > (555+offset3));
int t3_1_69 = rt3_1_69 * (4 + t3_0_139);
t3_1_69 += (1 - rt3_1_69) * t3_0_138;

int rt3_1_70 = (input[i] > (563+offset3));
int t3_1_70 = rt3_1_70 * (4 + t3_0_141);
t3_1_70 += (1 - rt3_1_70) * t3_0_140;

int rt3_1_71 = (input[i] > (571+offset3));
int t3_1_71 = rt3_1_71 * (4 + t3_0_143);
t3_1_71 += (1 - rt3_1_71) * t3_0_142;

int rt3_1_72 = (input[i] > (579+offset3));
int t3_1_72 = rt3_1_72 * (4 + t3_0_145);
t3_1_72 += (1 - rt3_1_72) * t3_0_144;

int rt3_1_73 = (input[i] > (587+offset3));
int t3_1_73 = rt3_1_73 * (4 + t3_0_147);
t3_1_73 += (1 - rt3_1_73) * t3_0_146;

int rt3_1_74 = (input[i] > (595+offset3));
int t3_1_74 = rt3_1_74 * (4 + t3_0_149);
t3_1_74 += (1 - rt3_1_74) * t3_0_148;

int rt3_1_75 = (input[i] > (603+offset3));
int t3_1_75 = rt3_1_75 * (4 + t3_0_151);
t3_1_75 += (1 - rt3_1_75) * t3_0_150;

int rt3_1_76 = (input[i] > (611+offset3));
int t3_1_76 = rt3_1_76 * (4 + t3_0_153);
t3_1_76 += (1 - rt3_1_76) * t3_0_152;

int rt3_1_77 = (input[i] > (619+offset3));
int t3_1_77 = rt3_1_77 * (4 + t3_0_155);
t3_1_77 += (1 - rt3_1_77) * t3_0_154;

int rt3_1_78 = (input[i] > (627+offset3));
int t3_1_78 = rt3_1_78 * (4 + t3_0_157);
t3_1_78 += (1 - rt3_1_78) * t3_0_156;

int rt3_1_79 = (input[i] > (635+offset3));
int t3_1_79 = rt3_1_79 * (4 + t3_0_159);
t3_1_79 += (1 - rt3_1_79) * t3_0_158;

int rt3_1_80 = (input[i] > (643+offset3));
int t3_1_80 = rt3_1_80 * (4 + t3_0_161);
t3_1_80 += (1 - rt3_1_80) * t3_0_160;

int rt3_1_81 = (input[i] > (651+offset3));
int t3_1_81 = rt3_1_81 * (4 + t3_0_163);
t3_1_81 += (1 - rt3_1_81) * t3_0_162;

int rt3_1_82 = (input[i] > (659+offset3));
int t3_1_82 = rt3_1_82 * (4 + t3_0_165);
t3_1_82 += (1 - rt3_1_82) * t3_0_164;

int rt3_1_83 = (input[i] > (667+offset3));
int t3_1_83 = rt3_1_83 * (4 + t3_0_167);
t3_1_83 += (1 - rt3_1_83) * t3_0_166;

int rt3_1_84 = (input[i] > (675+offset3));
int t3_1_84 = rt3_1_84 * (4 + t3_0_169);
t3_1_84 += (1 - rt3_1_84) * t3_0_168;

int rt3_1_85 = (input[i] > (683+offset3));
int t3_1_85 = rt3_1_85 * (4 + t3_0_171);
t3_1_85 += (1 - rt3_1_85) * t3_0_170;

int rt3_1_86 = (input[i] > (691+offset3));
int t3_1_86 = rt3_1_86 * (4 + t3_0_173);
t3_1_86 += (1 - rt3_1_86) * t3_0_172;

int rt3_1_87 = (input[i] > (699+offset3));
int t3_1_87 = rt3_1_87 * (4 + t3_0_175);
t3_1_87 += (1 - rt3_1_87) * t3_0_174;

int rt3_1_88 = (input[i] > (707+offset3));
int t3_1_88 = rt3_1_88 * (4 + t3_0_177);
t3_1_88 += (1 - rt3_1_88) * t3_0_176;

int rt3_1_89 = (input[i] > (715+offset3));
int t3_1_89 = rt3_1_89 * (4 + t3_0_179);
t3_1_89 += (1 - rt3_1_89) * t3_0_178;

int rt3_1_90 = (input[i] > (723+offset3));
int t3_1_90 = rt3_1_90 * (4 + t3_0_181);
t3_1_90 += (1 - rt3_1_90) * t3_0_180;

int rt3_1_91 = (input[i] > (731+offset3));
int t3_1_91 = rt3_1_91 * (4 + t3_0_183);
t3_1_91 += (1 - rt3_1_91) * t3_0_182;

int rt3_1_92 = (input[i] > (739+offset3));
int t3_1_92 = rt3_1_92 * (4 + t3_0_185);
t3_1_92 += (1 - rt3_1_92) * t3_0_184;

int rt3_1_93 = (input[i] > (747+offset3));
int t3_1_93 = rt3_1_93 * (4 + t3_0_187);
t3_1_93 += (1 - rt3_1_93) * t3_0_186;

int rt3_1_94 = (input[i] > (755+offset3));
int t3_1_94 = rt3_1_94 * (4 + t3_0_189);
t3_1_94 += (1 - rt3_1_94) * t3_0_188;

int rt3_1_95 = (input[i] > (763+offset3));
int t3_1_95 = rt3_1_95 * (4 + t3_0_191);
t3_1_95 += (1 - rt3_1_95) * t3_0_190;

int rt3_1_96 = (input[i] > (771+offset3));
int t3_1_96 = rt3_1_96 * (4 + t3_0_193);
t3_1_96 += (1 - rt3_1_96) * t3_0_192;

int rt3_1_97 = (input[i] > (779+offset3));
int t3_1_97 = rt3_1_97 * (4 + t3_0_195);
t3_1_97 += (1 - rt3_1_97) * t3_0_194;

int rt3_1_98 = (input[i] > (787+offset3));
int t3_1_98 = rt3_1_98 * (4 + t3_0_197);
t3_1_98 += (1 - rt3_1_98) * t3_0_196;

int rt3_1_99 = (input[i] > (795+offset3));
int t3_1_99 = rt3_1_99 * (4 + t3_0_199);
t3_1_99 += (1 - rt3_1_99) * t3_0_198;

int rt3_1_100 = (input[i] > (803+offset3));
int t3_1_100 = rt3_1_100 * (4 + t3_0_201);
t3_1_100 += (1 - rt3_1_100) * t3_0_200;

int rt3_1_101 = (input[i] > (811+offset3));
int t3_1_101 = rt3_1_101 * (4 + t3_0_203);
t3_1_101 += (1 - rt3_1_101) * t3_0_202;

int rt3_1_102 = (input[i] > (819+offset3));
int t3_1_102 = rt3_1_102 * (4 + t3_0_205);
t3_1_102 += (1 - rt3_1_102) * t3_0_204;

int rt3_1_103 = (input[i] > (827+offset3));
int t3_1_103 = rt3_1_103 * (4 + t3_0_207);
t3_1_103 += (1 - rt3_1_103) * t3_0_206;

int rt3_1_104 = (input[i] > (835+offset3));
int t3_1_104 = rt3_1_104 * (4 + t3_0_209);
t3_1_104 += (1 - rt3_1_104) * t3_0_208;

int rt3_1_105 = (input[i] > (843+offset3));
int t3_1_105 = rt3_1_105 * (4 + t3_0_211);
t3_1_105 += (1 - rt3_1_105) * t3_0_210;

int rt3_1_106 = (input[i] > (851+offset3));
int t3_1_106 = rt3_1_106 * (4 + t3_0_213);
t3_1_106 += (1 - rt3_1_106) * t3_0_212;

int rt3_1_107 = (input[i] > (859+offset3));
int t3_1_107 = rt3_1_107 * (4 + t3_0_215);
t3_1_107 += (1 - rt3_1_107) * t3_0_214;

int rt3_1_108 = (input[i] > (867+offset3));
int t3_1_108 = rt3_1_108 * (4 + t3_0_217);
t3_1_108 += (1 - rt3_1_108) * t3_0_216;

int rt3_1_109 = (input[i] > (875+offset3));
int t3_1_109 = rt3_1_109 * (4 + t3_0_219);
t3_1_109 += (1 - rt3_1_109) * t3_0_218;

int rt3_1_110 = (input[i] > (883+offset3));
int t3_1_110 = rt3_1_110 * (4 + t3_0_221);
t3_1_110 += (1 - rt3_1_110) * t3_0_220;

int rt3_1_111 = (input[i] > (891+offset3));
int t3_1_111 = rt3_1_111 * (4 + t3_0_223);
t3_1_111 += (1 - rt3_1_111) * t3_0_222;

int rt3_1_112 = (input[i] > (899+offset3));
int t3_1_112 = rt3_1_112 * (4 + t3_0_225);
t3_1_112 += (1 - rt3_1_112) * t3_0_224;

int rt3_1_113 = (input[i] > (907+offset3));
int t3_1_113 = rt3_1_113 * (4 + t3_0_227);
t3_1_113 += (1 - rt3_1_113) * t3_0_226;

int rt3_1_114 = (input[i] > (915+offset3));
int t3_1_114 = rt3_1_114 * (4 + t3_0_229);
t3_1_114 += (1 - rt3_1_114) * t3_0_228;

int rt3_1_115 = (input[i] > (923+offset3));
int t3_1_115 = rt3_1_115 * (4 + t3_0_231);
t3_1_115 += (1 - rt3_1_115) * t3_0_230;

int rt3_1_116 = (input[i] > (931+offset3));
int t3_1_116 = rt3_1_116 * (4 + t3_0_233);
t3_1_116 += (1 - rt3_1_116) * t3_0_232;

int rt3_1_117 = (input[i] > (939+offset3));
int t3_1_117 = rt3_1_117 * (4 + t3_0_235);
t3_1_117 += (1 - rt3_1_117) * t3_0_234;

int rt3_1_118 = (input[i] > (947+offset3));
int t3_1_118 = rt3_1_118 * (4 + t3_0_237);
t3_1_118 += (1 - rt3_1_118) * t3_0_236;

int rt3_1_119 = (input[i] > (955+offset3));
int t3_1_119 = rt3_1_119 * (4 + t3_0_239);
t3_1_119 += (1 - rt3_1_119) * t3_0_238;

int rt3_1_120 = (input[i] > (963+offset3));
int t3_1_120 = rt3_1_120 * (4 + t3_0_241);
t3_1_120 += (1 - rt3_1_120) * t3_0_240;

int rt3_1_121 = (input[i] > (971+offset3));
int t3_1_121 = rt3_1_121 * (4 + t3_0_243);
t3_1_121 += (1 - rt3_1_121) * t3_0_242;

int rt3_1_122 = (input[i] > (979+offset3));
int t3_1_122 = rt3_1_122 * (4 + t3_0_245);
t3_1_122 += (1 - rt3_1_122) * t3_0_244;

int rt3_1_123 = (input[i] > (987+offset3));
int t3_1_123 = rt3_1_123 * (4 + t3_0_247);
t3_1_123 += (1 - rt3_1_123) * t3_0_246;

int rt3_1_124 = (input[i] > (995+offset3));
int t3_1_124 = rt3_1_124 * (4 + t3_0_249);
t3_1_124 += (1 - rt3_1_124) * t3_0_248;

int rt3_1_125 = (input[i] > (1003+offset3));
int t3_1_125 = rt3_1_125 * (4 + t3_0_251);
t3_1_125 += (1 - rt3_1_125) * t3_0_250;

int rt3_1_126 = (input[i] > (1011+offset3));
int t3_1_126 = rt3_1_126 * (4 + t3_0_253);
t3_1_126 += (1 - rt3_1_126) * t3_0_252;

int rt3_1_127 = (input[i] > (1019+offset3));
int t3_1_127 = rt3_1_127 * (4 + t3_0_255);
t3_1_127 += (1 - rt3_1_127) * t3_0_254;

int rt3_2_0 = (input[i] > (7+offset3));
int t3_2_0 = rt3_2_0 * (8 + t3_1_1);
t3_2_0 += (1 - rt3_2_0) * t3_1_0;

int rt3_2_1 = (input[i] > (23+offset3));
int t3_2_1 = rt3_2_1 * (8 + t3_1_3);
t3_2_1 += (1 - rt3_2_1) * t3_1_2;

int rt3_2_2 = (input[i] > (39+offset3));
int t3_2_2 = rt3_2_2 * (8 + t3_1_5);
t3_2_2 += (1 - rt3_2_2) * t3_1_4;

int rt3_2_3 = (input[i] > (55+offset3));
int t3_2_3 = rt3_2_3 * (8 + t3_1_7);
t3_2_3 += (1 - rt3_2_3) * t3_1_6;

int rt3_2_4 = (input[i] > (71+offset3));
int t3_2_4 = rt3_2_4 * (8 + t3_1_9);
t3_2_4 += (1 - rt3_2_4) * t3_1_8;

int rt3_2_5 = (input[i] > (87+offset3));
int t3_2_5 = rt3_2_5 * (8 + t3_1_11);
t3_2_5 += (1 - rt3_2_5) * t3_1_10;

int rt3_2_6 = (input[i] > (103+offset3));
int t3_2_6 = rt3_2_6 * (8 + t3_1_13);
t3_2_6 += (1 - rt3_2_6) * t3_1_12;

int rt3_2_7 = (input[i] > (119+offset3));
int t3_2_7 = rt3_2_7 * (8 + t3_1_15);
t3_2_7 += (1 - rt3_2_7) * t3_1_14;

int rt3_2_8 = (input[i] > (135+offset3));
int t3_2_8 = rt3_2_8 * (8 + t3_1_17);
t3_2_8 += (1 - rt3_2_8) * t3_1_16;

int rt3_2_9 = (input[i] > (151+offset3));
int t3_2_9 = rt3_2_9 * (8 + t3_1_19);
t3_2_9 += (1 - rt3_2_9) * t3_1_18;

int rt3_2_10 = (input[i] > (167+offset3));
int t3_2_10 = rt3_2_10 * (8 + t3_1_21);
t3_2_10 += (1 - rt3_2_10) * t3_1_20;

int rt3_2_11 = (input[i] > (183+offset3));
int t3_2_11 = rt3_2_11 * (8 + t3_1_23);
t3_2_11 += (1 - rt3_2_11) * t3_1_22;

int rt3_2_12 = (input[i] > (199+offset3));
int t3_2_12 = rt3_2_12 * (8 + t3_1_25);
t3_2_12 += (1 - rt3_2_12) * t3_1_24;

int rt3_2_13 = (input[i] > (215+offset3));
int t3_2_13 = rt3_2_13 * (8 + t3_1_27);
t3_2_13 += (1 - rt3_2_13) * t3_1_26;

int rt3_2_14 = (input[i] > (231+offset3));
int t3_2_14 = rt3_2_14 * (8 + t3_1_29);
t3_2_14 += (1 - rt3_2_14) * t3_1_28;

int rt3_2_15 = (input[i] > (247+offset3));
int t3_2_15 = rt3_2_15 * (8 + t3_1_31);
t3_2_15 += (1 - rt3_2_15) * t3_1_30;

int rt3_2_16 = (input[i] > (263+offset3));
int t3_2_16 = rt3_2_16 * (8 + t3_1_33);
t3_2_16 += (1 - rt3_2_16) * t3_1_32;

int rt3_2_17 = (input[i] > (279+offset3));
int t3_2_17 = rt3_2_17 * (8 + t3_1_35);
t3_2_17 += (1 - rt3_2_17) * t3_1_34;

int rt3_2_18 = (input[i] > (295+offset3));
int t3_2_18 = rt3_2_18 * (8 + t3_1_37);
t3_2_18 += (1 - rt3_2_18) * t3_1_36;

int rt3_2_19 = (input[i] > (311+offset3));
int t3_2_19 = rt3_2_19 * (8 + t3_1_39);
t3_2_19 += (1 - rt3_2_19) * t3_1_38;

int rt3_2_20 = (input[i] > (327+offset3));
int t3_2_20 = rt3_2_20 * (8 + t3_1_41);
t3_2_20 += (1 - rt3_2_20) * t3_1_40;

int rt3_2_21 = (input[i] > (343+offset3));
int t3_2_21 = rt3_2_21 * (8 + t3_1_43);
t3_2_21 += (1 - rt3_2_21) * t3_1_42;

int rt3_2_22 = (input[i] > (359+offset3));
int t3_2_22 = rt3_2_22 * (8 + t3_1_45);
t3_2_22 += (1 - rt3_2_22) * t3_1_44;

int rt3_2_23 = (input[i] > (375+offset3));
int t3_2_23 = rt3_2_23 * (8 + t3_1_47);
t3_2_23 += (1 - rt3_2_23) * t3_1_46;

int rt3_2_24 = (input[i] > (391+offset3));
int t3_2_24 = rt3_2_24 * (8 + t3_1_49);
t3_2_24 += (1 - rt3_2_24) * t3_1_48;

int rt3_2_25 = (input[i] > (407+offset3));
int t3_2_25 = rt3_2_25 * (8 + t3_1_51);
t3_2_25 += (1 - rt3_2_25) * t3_1_50;

int rt3_2_26 = (input[i] > (423+offset3));
int t3_2_26 = rt3_2_26 * (8 + t3_1_53);
t3_2_26 += (1 - rt3_2_26) * t3_1_52;

int rt3_2_27 = (input[i] > (439+offset3));
int t3_2_27 = rt3_2_27 * (8 + t3_1_55);
t3_2_27 += (1 - rt3_2_27) * t3_1_54;

int rt3_2_28 = (input[i] > (455+offset3));
int t3_2_28 = rt3_2_28 * (8 + t3_1_57);
t3_2_28 += (1 - rt3_2_28) * t3_1_56;

int rt3_2_29 = (input[i] > (471+offset3));
int t3_2_29 = rt3_2_29 * (8 + t3_1_59);
t3_2_29 += (1 - rt3_2_29) * t3_1_58;

int rt3_2_30 = (input[i] > (487+offset3));
int t3_2_30 = rt3_2_30 * (8 + t3_1_61);
t3_2_30 += (1 - rt3_2_30) * t3_1_60;

int rt3_2_31 = (input[i] > (503+offset3));
int t3_2_31 = rt3_2_31 * (8 + t3_1_63);
t3_2_31 += (1 - rt3_2_31) * t3_1_62;

int rt3_2_32 = (input[i] > (519+offset3));
int t3_2_32 = rt3_2_32 * (8 + t3_1_65);
t3_2_32 += (1 - rt3_2_32) * t3_1_64;

int rt3_2_33 = (input[i] > (535+offset3));
int t3_2_33 = rt3_2_33 * (8 + t3_1_67);
t3_2_33 += (1 - rt3_2_33) * t3_1_66;

int rt3_2_34 = (input[i] > (551+offset3));
int t3_2_34 = rt3_2_34 * (8 + t3_1_69);
t3_2_34 += (1 - rt3_2_34) * t3_1_68;

int rt3_2_35 = (input[i] > (567+offset3));
int t3_2_35 = rt3_2_35 * (8 + t3_1_71);
t3_2_35 += (1 - rt3_2_35) * t3_1_70;

int rt3_2_36 = (input[i] > (583+offset3));
int t3_2_36 = rt3_2_36 * (8 + t3_1_73);
t3_2_36 += (1 - rt3_2_36) * t3_1_72;

int rt3_2_37 = (input[i] > (599+offset3));
int t3_2_37 = rt3_2_37 * (8 + t3_1_75);
t3_2_37 += (1 - rt3_2_37) * t3_1_74;

int rt3_2_38 = (input[i] > (615+offset3));
int t3_2_38 = rt3_2_38 * (8 + t3_1_77);
t3_2_38 += (1 - rt3_2_38) * t3_1_76;

int rt3_2_39 = (input[i] > (631+offset3));
int t3_2_39 = rt3_2_39 * (8 + t3_1_79);
t3_2_39 += (1 - rt3_2_39) * t3_1_78;

int rt3_2_40 = (input[i] > (647+offset3));
int t3_2_40 = rt3_2_40 * (8 + t3_1_81);
t3_2_40 += (1 - rt3_2_40) * t3_1_80;

int rt3_2_41 = (input[i] > (663+offset3));
int t3_2_41 = rt3_2_41 * (8 + t3_1_83);
t3_2_41 += (1 - rt3_2_41) * t3_1_82;

int rt3_2_42 = (input[i] > (679+offset3));
int t3_2_42 = rt3_2_42 * (8 + t3_1_85);
t3_2_42 += (1 - rt3_2_42) * t3_1_84;

int rt3_2_43 = (input[i] > (695+offset3));
int t3_2_43 = rt3_2_43 * (8 + t3_1_87);
t3_2_43 += (1 - rt3_2_43) * t3_1_86;

int rt3_2_44 = (input[i] > (711+offset3));
int t3_2_44 = rt3_2_44 * (8 + t3_1_89);
t3_2_44 += (1 - rt3_2_44) * t3_1_88;

int rt3_2_45 = (input[i] > (727+offset3));
int t3_2_45 = rt3_2_45 * (8 + t3_1_91);
t3_2_45 += (1 - rt3_2_45) * t3_1_90;

int rt3_2_46 = (input[i] > (743+offset3));
int t3_2_46 = rt3_2_46 * (8 + t3_1_93);
t3_2_46 += (1 - rt3_2_46) * t3_1_92;

int rt3_2_47 = (input[i] > (759+offset3));
int t3_2_47 = rt3_2_47 * (8 + t3_1_95);
t3_2_47 += (1 - rt3_2_47) * t3_1_94;

int rt3_2_48 = (input[i] > (775+offset3));
int t3_2_48 = rt3_2_48 * (8 + t3_1_97);
t3_2_48 += (1 - rt3_2_48) * t3_1_96;

int rt3_2_49 = (input[i] > (791+offset3));
int t3_2_49 = rt3_2_49 * (8 + t3_1_99);
t3_2_49 += (1 - rt3_2_49) * t3_1_98;

int rt3_2_50 = (input[i] > (807+offset3));
int t3_2_50 = rt3_2_50 * (8 + t3_1_101);
t3_2_50 += (1 - rt3_2_50) * t3_1_100;

int rt3_2_51 = (input[i] > (823+offset3));
int t3_2_51 = rt3_2_51 * (8 + t3_1_103);
t3_2_51 += (1 - rt3_2_51) * t3_1_102;

int rt3_2_52 = (input[i] > (839+offset3));
int t3_2_52 = rt3_2_52 * (8 + t3_1_105);
t3_2_52 += (1 - rt3_2_52) * t3_1_104;

int rt3_2_53 = (input[i] > (855+offset3));
int t3_2_53 = rt3_2_53 * (8 + t3_1_107);
t3_2_53 += (1 - rt3_2_53) * t3_1_106;

int rt3_2_54 = (input[i] > (871+offset3));
int t3_2_54 = rt3_2_54 * (8 + t3_1_109);
t3_2_54 += (1 - rt3_2_54) * t3_1_108;

int rt3_2_55 = (input[i] > (887+offset3));
int t3_2_55 = rt3_2_55 * (8 + t3_1_111);
t3_2_55 += (1 - rt3_2_55) * t3_1_110;

int rt3_2_56 = (input[i] > (903+offset3));
int t3_2_56 = rt3_2_56 * (8 + t3_1_113);
t3_2_56 += (1 - rt3_2_56) * t3_1_112;

int rt3_2_57 = (input[i] > (919+offset3));
int t3_2_57 = rt3_2_57 * (8 + t3_1_115);
t3_2_57 += (1 - rt3_2_57) * t3_1_114;

int rt3_2_58 = (input[i] > (935+offset3));
int t3_2_58 = rt3_2_58 * (8 + t3_1_117);
t3_2_58 += (1 - rt3_2_58) * t3_1_116;

int rt3_2_59 = (input[i] > (951+offset3));
int t3_2_59 = rt3_2_59 * (8 + t3_1_119);
t3_2_59 += (1 - rt3_2_59) * t3_1_118;

int rt3_2_60 = (input[i] > (967+offset3));
int t3_2_60 = rt3_2_60 * (8 + t3_1_121);
t3_2_60 += (1 - rt3_2_60) * t3_1_120;

int rt3_2_61 = (input[i] > (983+offset3));
int t3_2_61 = rt3_2_61 * (8 + t3_1_123);
t3_2_61 += (1 - rt3_2_61) * t3_1_122;

int rt3_2_62 = (input[i] > (999+offset3));
int t3_2_62 = rt3_2_62 * (8 + t3_1_125);
t3_2_62 += (1 - rt3_2_62) * t3_1_124;

int rt3_2_63 = (input[i] > (1015+offset3));
int t3_2_63 = rt3_2_63 * (8 + t3_1_127);
t3_2_63 += (1 - rt3_2_63) * t3_1_126;

int rt3_3_0 = (input[i] > (15+offset3));
int t3_3_0 = rt3_3_0 * (16 + t3_2_1);
t3_3_0 += (1 - rt3_3_0) * t3_2_0;

int rt3_3_1 = (input[i] > (47+offset3));
int t3_3_1 = rt3_3_1 * (16 + t3_2_3);
t3_3_1 += (1 - rt3_3_1) * t3_2_2;

int rt3_3_2 = (input[i] > (79+offset3));
int t3_3_2 = rt3_3_2 * (16 + t3_2_5);
t3_3_2 += (1 - rt3_3_2) * t3_2_4;

int rt3_3_3 = (input[i] > (111+offset3));
int t3_3_3 = rt3_3_3 * (16 + t3_2_7);
t3_3_3 += (1 - rt3_3_3) * t3_2_6;

int rt3_3_4 = (input[i] > (143+offset3));
int t3_3_4 = rt3_3_4 * (16 + t3_2_9);
t3_3_4 += (1 - rt3_3_4) * t3_2_8;

int rt3_3_5 = (input[i] > (175+offset3));
int t3_3_5 = rt3_3_5 * (16 + t3_2_11);
t3_3_5 += (1 - rt3_3_5) * t3_2_10;

int rt3_3_6 = (input[i] > (207+offset3));
int t3_3_6 = rt3_3_6 * (16 + t3_2_13);
t3_3_6 += (1 - rt3_3_6) * t3_2_12;

int rt3_3_7 = (input[i] > (239+offset3));
int t3_3_7 = rt3_3_7 * (16 + t3_2_15);
t3_3_7 += (1 - rt3_3_7) * t3_2_14;

int rt3_3_8 = (input[i] > (271+offset3));
int t3_3_8 = rt3_3_8 * (16 + t3_2_17);
t3_3_8 += (1 - rt3_3_8) * t3_2_16;

int rt3_3_9 = (input[i] > (303+offset3));
int t3_3_9 = rt3_3_9 * (16 + t3_2_19);
t3_3_9 += (1 - rt3_3_9) * t3_2_18;

int rt3_3_10 = (input[i] > (335+offset3));
int t3_3_10 = rt3_3_10 * (16 + t3_2_21);
t3_3_10 += (1 - rt3_3_10) * t3_2_20;

int rt3_3_11 = (input[i] > (367+offset3));
int t3_3_11 = rt3_3_11 * (16 + t3_2_23);
t3_3_11 += (1 - rt3_3_11) * t3_2_22;

int rt3_3_12 = (input[i] > (399+offset3));
int t3_3_12 = rt3_3_12 * (16 + t3_2_25);
t3_3_12 += (1 - rt3_3_12) * t3_2_24;

int rt3_3_13 = (input[i] > (431+offset3));
int t3_3_13 = rt3_3_13 * (16 + t3_2_27);
t3_3_13 += (1 - rt3_3_13) * t3_2_26;

int rt3_3_14 = (input[i] > (463+offset3));
int t3_3_14 = rt3_3_14 * (16 + t3_2_29);
t3_3_14 += (1 - rt3_3_14) * t3_2_28;

int rt3_3_15 = (input[i] > (495+offset3));
int t3_3_15 = rt3_3_15 * (16 + t3_2_31);
t3_3_15 += (1 - rt3_3_15) * t3_2_30;

int rt3_3_16 = (input[i] > (527+offset3));
int t3_3_16 = rt3_3_16 * (16 + t3_2_33);
t3_3_16 += (1 - rt3_3_16) * t3_2_32;

int rt3_3_17 = (input[i] > (559+offset3));
int t3_3_17 = rt3_3_17 * (16 + t3_2_35);
t3_3_17 += (1 - rt3_3_17) * t3_2_34;

int rt3_3_18 = (input[i] > (591+offset3));
int t3_3_18 = rt3_3_18 * (16 + t3_2_37);
t3_3_18 += (1 - rt3_3_18) * t3_2_36;

int rt3_3_19 = (input[i] > (623+offset3));
int t3_3_19 = rt3_3_19 * (16 + t3_2_39);
t3_3_19 += (1 - rt3_3_19) * t3_2_38;

int rt3_3_20 = (input[i] > (655+offset3));
int t3_3_20 = rt3_3_20 * (16 + t3_2_41);
t3_3_20 += (1 - rt3_3_20) * t3_2_40;

int rt3_3_21 = (input[i] > (687+offset3));
int t3_3_21 = rt3_3_21 * (16 + t3_2_43);
t3_3_21 += (1 - rt3_3_21) * t3_2_42;

int rt3_3_22 = (input[i] > (719+offset3));
int t3_3_22 = rt3_3_22 * (16 + t3_2_45);
t3_3_22 += (1 - rt3_3_22) * t3_2_44;

int rt3_3_23 = (input[i] > (751+offset3));
int t3_3_23 = rt3_3_23 * (16 + t3_2_47);
t3_3_23 += (1 - rt3_3_23) * t3_2_46;

int rt3_3_24 = (input[i] > (783+offset3));
int t3_3_24 = rt3_3_24 * (16 + t3_2_49);
t3_3_24 += (1 - rt3_3_24) * t3_2_48;

int rt3_3_25 = (input[i] > (815+offset3));
int t3_3_25 = rt3_3_25 * (16 + t3_2_51);
t3_3_25 += (1 - rt3_3_25) * t3_2_50;

int rt3_3_26 = (input[i] > (847+offset3));
int t3_3_26 = rt3_3_26 * (16 + t3_2_53);
t3_3_26 += (1 - rt3_3_26) * t3_2_52;

int rt3_3_27 = (input[i] > (879+offset3));
int t3_3_27 = rt3_3_27 * (16 + t3_2_55);
t3_3_27 += (1 - rt3_3_27) * t3_2_54;

int rt3_3_28 = (input[i] > (911+offset3));
int t3_3_28 = rt3_3_28 * (16 + t3_2_57);
t3_3_28 += (1 - rt3_3_28) * t3_2_56;

int rt3_3_29 = (input[i] > (943+offset3));
int t3_3_29 = rt3_3_29 * (16 + t3_2_59);
t3_3_29 += (1 - rt3_3_29) * t3_2_58;

int rt3_3_30 = (input[i] > (975+offset3));
int t3_3_30 = rt3_3_30 * (16 + t3_2_61);
t3_3_30 += (1 - rt3_3_30) * t3_2_60;

int rt3_3_31 = (input[i] > (1007+offset3));
int t3_3_31 = rt3_3_31 * (16 + t3_2_63);
t3_3_31 += (1 - rt3_3_31) * t3_2_62;

int rt3_4_0 = (input[i] > (31+offset3));
int t3_4_0 = rt3_4_0 * (32 + t3_3_1);
t3_4_0 += (1 - rt3_4_0) * t3_3_0;

int rt3_4_1 = (input[i] > (95+offset3));
int t3_4_1 = rt3_4_1 * (32 + t3_3_3);
t3_4_1 += (1 - rt3_4_1) * t3_3_2;

int rt3_4_2 = (input[i] > (159+offset3));
int t3_4_2 = rt3_4_2 * (32 + t3_3_5);
t3_4_2 += (1 - rt3_4_2) * t3_3_4;

int rt3_4_3 = (input[i] > (223+offset3));
int t3_4_3 = rt3_4_3 * (32 + t3_3_7);
t3_4_3 += (1 - rt3_4_3) * t3_3_6;

int rt3_4_4 = (input[i] > (287+offset3));
int t3_4_4 = rt3_4_4 * (32 + t3_3_9);
t3_4_4 += (1 - rt3_4_4) * t3_3_8;

int rt3_4_5 = (input[i] > (351+offset3));
int t3_4_5 = rt3_4_5 * (32 + t3_3_11);
t3_4_5 += (1 - rt3_4_5) * t3_3_10;

int rt3_4_6 = (input[i] > (415+offset3));
int t3_4_6 = rt3_4_6 * (32 + t3_3_13);
t3_4_6 += (1 - rt3_4_6) * t3_3_12;

int rt3_4_7 = (input[i] > (479+offset3));
int t3_4_7 = rt3_4_7 * (32 + t3_3_15);
t3_4_7 += (1 - rt3_4_7) * t3_3_14;

int rt3_4_8 = (input[i] > (543+offset3));
int t3_4_8 = rt3_4_8 * (32 + t3_3_17);
t3_4_8 += (1 - rt3_4_8) * t3_3_16;

int rt3_4_9 = (input[i] > (607+offset3));
int t3_4_9 = rt3_4_9 * (32 + t3_3_19);
t3_4_9 += (1 - rt3_4_9) * t3_3_18;

int rt3_4_10 = (input[i] > (671+offset3));
int t3_4_10 = rt3_4_10 * (32 + t3_3_21);
t3_4_10 += (1 - rt3_4_10) * t3_3_20;

int rt3_4_11 = (input[i] > (735+offset3));
int t3_4_11 = rt3_4_11 * (32 + t3_3_23);
t3_4_11 += (1 - rt3_4_11) * t3_3_22;

int rt3_4_12 = (input[i] > (799+offset3));
int t3_4_12 = rt3_4_12 * (32 + t3_3_25);
t3_4_12 += (1 - rt3_4_12) * t3_3_24;

int rt3_4_13 = (input[i] > (863+offset3));
int t3_4_13 = rt3_4_13 * (32 + t3_3_27);
t3_4_13 += (1 - rt3_4_13) * t3_3_26;

int rt3_4_14 = (input[i] > (927+offset3));
int t3_4_14 = rt3_4_14 * (32 + t3_3_29);
t3_4_14 += (1 - rt3_4_14) * t3_3_28;

int rt3_4_15 = (input[i] > (991+offset3));
int t3_4_15 = rt3_4_15 * (32 + t3_3_31);
t3_4_15 += (1 - rt3_4_15) * t3_3_30;

int rt3_5_0 = (input[i] > (63+offset3));
int t3_5_0 = rt3_5_0 * (64 + t3_4_1);
t3_5_0 += (1 - rt3_5_0) * t3_4_0;

int rt3_5_1 = (input[i] > (191+offset3));
int t3_5_1 = rt3_5_1 * (64 + t3_4_3);
t3_5_1 += (1 - rt3_5_1) * t3_4_2;

int rt3_5_2 = (input[i] > (319+offset3));
int t3_5_2 = rt3_5_2 * (64 + t3_4_5);
t3_5_2 += (1 - rt3_5_2) * t3_4_4;

int rt3_5_3 = (input[i] > (447+offset3));
int t3_5_3 = rt3_5_3 * (64 + t3_4_7);
t3_5_3 += (1 - rt3_5_3) * t3_4_6;

int rt3_5_4 = (input[i] > (575+offset3));
int t3_5_4 = rt3_5_4 * (64 + t3_4_9);
t3_5_4 += (1 - rt3_5_4) * t3_4_8;

int rt3_5_5 = (input[i] > (703+offset3));
int t3_5_5 = rt3_5_5 * (64 + t3_4_11);
t3_5_5 += (1 - rt3_5_5) * t3_4_10;

int rt3_5_6 = (input[i] > (831+offset3));
int t3_5_6 = rt3_5_6 * (64 + t3_4_13);
t3_5_6 += (1 - rt3_5_6) * t3_4_12;

int rt3_5_7 = (input[i] > (959+offset3));
int t3_5_7 = rt3_5_7 * (64 + t3_4_15);
t3_5_7 += (1 - rt3_5_7) * t3_4_14;

int rt3_6_0 = (input[i] > (127+offset3));
int t3_6_0 = rt3_6_0 * (128 + t3_5_1);
t3_6_0 += (1 - rt3_6_0) * t3_5_0;

int rt3_6_1 = (input[i] > (383+offset3));
int t3_6_1 = rt3_6_1 * (128 + t3_5_3);
t3_6_1 += (1 - rt3_6_1) * t3_5_2;

int rt3_6_2 = (input[i] > (639+offset3));
int t3_6_2 = rt3_6_2 * (128 + t3_5_5);
t3_6_2 += (1 - rt3_6_2) * t3_5_4;

int rt3_6_3 = (input[i] > (895+offset3));
int t3_6_3 = rt3_6_3 * (128 + t3_5_7);
t3_6_3 += (1 - rt3_6_3) * t3_5_6;

int rt3_7_0 = (input[i] > (255+offset3));
int t3_7_0 = rt3_7_0 * (256 + t3_6_1);
t3_7_0 += (1 - rt3_7_0) * t3_6_0;

int rt3_7_1 = (input[i] > (767+offset3));
int t3_7_1 = rt3_7_1 * (256 + t3_6_3);
t3_7_1 += (1 - rt3_7_1) * t3_6_2;

int rt3_8_0 = (input[i] > (511+offset3));
int t3_8_0 = rt3_8_0 * (512 + t3_7_1);
t3_8_0 += (1 - rt3_8_0) * t3_7_0;

vR[i] = t0_8_0+ (t0_8_0 == t1_8_0)+ (t0_8_0 == t2_8_0)+ (t0_8_0 == t3_8_0);
	}
}

int main(int argc, char **argv)
{
	float elapsed_time;
	// set up device
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("[CUDA - Random Forest]: EXP %s\n", EXP_NAME);
	printf("[CUDA - Random Forest]: Using Device %d: %s\n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));

	// set up data size of vectors
	int nElem = N_ELEM;
	printf("[CUDA - Random Forest]: Vector Size %d\n", nElem);

	// malloc host memory
	size_t nBytes = nElem * sizeof(float);
	float *h_vR;
	h_vR = (float *)malloc(nBytes);
	float *h_input;
	h_input = (float *)malloc(nBytes);

	printf("[CUDA - Random Forest]: Start Reading Dataset\n");
	initData(h_input, nElem);
	printf("[CUDA - Random Forest]: Finished Reading Dataset\n");
	memset(h_vR, 0, nBytes);

	// malloc device global memory
	float *d_vR;
	CHECK(hipMalloc((float **)&d_vR, nBytes));
	float *d_input;
	CHECK(hipMalloc((float **)&d_input, nBytes));

	// transfer data from host to device
	CHECK(hipMemcpy(d_input, h_input, nBytes, hipMemcpyHostToDevice));

	// invoke kernel at host side
	int iLen = 512;
	dim3 block(iLen);
	dim3 grid((nElem + block.x - 1) / block.x);
	hipEvent_t start, stop;
	CHECK(hipEventCreate(&start));
	CHECK(hipEventCreate(&stop));

	// record start event
	CHECK(hipEventRecord(start, 0));
#if defined(_2TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, d_vR);
#elif defined(_3TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, d_vR);
#elif defined(_4TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, d_vR);
#else
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, d_vR);
#endif
	CHECK(hipEventRecord(stop, 0));
	CHECK(hipEventSynchronize(stop));
	hipDeviceSynchronize();

	// calculate elapsed time
	CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
	printf("[CUDA - Random Forest]: %s - execution time = %.6fms\n", EXP_NAME, elapsed_time);
	registerTime(elapsed_time);

	// copy kernel result back to host side
	CHECK(hipMemcpy(h_vR, d_vR, nBytes, hipMemcpyDeviceToHost));
	CHECK(hipGetLastError());

	//Verify the answers
	char flag = 1;
	for (int i = 0; i < nElem; i++)
	{
		if (i < 32)
		{
			printf("%.3f ", h_vR[i]);
		}
		if (h_input[i] != h_vR[i])
		{
			flag = 0;
			break;
		}
	}
	if (flag == 1)
	{
//		printf("Return correct");
	}
	else
	{
//		printf("Error!!");
	}
	printf("\n ");

	// free device memory
	hipFree(d_input);
	hipFree(d_vR);

	// free host memory
	free(h_input);
	free(h_vR);
	return (0);
}

void initData(float *ip, int size)
{
	for (int i = 0; i < size; i++)
	{
		ip[i] = (float)(0);
	}
	return;
}

void registerTime(float value)
{
	printf("[CUDA - Random Forest]: Time spent: %f.\n", value);
}
