#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>

#define N_ELEM 50000000

#define WARP_SIZE 32

#define CHECK(call)                                                \
	{                                                              \
		const hipError_t error = call;                            \
		if (error != hipSuccess)                                  \
		{                                                          \
			fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
			fprintf(stderr, "code: %d, reason: %s\n", error,       \
					hipGetErrorString(error));                    \
		}                                                          \
	}

void initData(float *ip, int size);
void registerTime(float value);

#define _MOD 1024
#define _2TREE
#define EXP_NAME "_RF_CALC_2TREE_10LV_1INPUT_50000000"

// RF_with_if

__global__ void RF_kernel(float *input, const int n, const int offset0, const int offset1, float *vR)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
int rt0_0_0 = (input[i] > (1+offset0));
int t0_0_0 = rt0_0_0 * (2 + (input[i] > (2+offset0)));
t0_0_0 += (1 - rt0_0_0) * (input[i] > (0+offset0));

int rt0_0_1 = (input[i] > (5+offset0));
int t0_0_1 = rt0_0_1 * (2 + (input[i] > (6+offset0)));
t0_0_1 += (1 - rt0_0_1) * (input[i] > (4+offset0));

int rt0_0_2 = (input[i] > (9+offset0));
int t0_0_2 = rt0_0_2 * (2 + (input[i] > (10+offset0)));
t0_0_2 += (1 - rt0_0_2) * (input[i] > (8+offset0));

int rt0_0_3 = (input[i] > (13+offset0));
int t0_0_3 = rt0_0_3 * (2 + (input[i] > (14+offset0)));
t0_0_3 += (1 - rt0_0_3) * (input[i] > (12+offset0));

int rt0_0_4 = (input[i] > (17+offset0));
int t0_0_4 = rt0_0_4 * (2 + (input[i] > (18+offset0)));
t0_0_4 += (1 - rt0_0_4) * (input[i] > (16+offset0));

int rt0_0_5 = (input[i] > (21+offset0));
int t0_0_5 = rt0_0_5 * (2 + (input[i] > (22+offset0)));
t0_0_5 += (1 - rt0_0_5) * (input[i] > (20+offset0));

int rt0_0_6 = (input[i] > (25+offset0));
int t0_0_6 = rt0_0_6 * (2 + (input[i] > (26+offset0)));
t0_0_6 += (1 - rt0_0_6) * (input[i] > (24+offset0));

int rt0_0_7 = (input[i] > (29+offset0));
int t0_0_7 = rt0_0_7 * (2 + (input[i] > (30+offset0)));
t0_0_7 += (1 - rt0_0_7) * (input[i] > (28+offset0));

int rt0_0_8 = (input[i] > (33+offset0));
int t0_0_8 = rt0_0_8 * (2 + (input[i] > (34+offset0)));
t0_0_8 += (1 - rt0_0_8) * (input[i] > (32+offset0));

int rt0_0_9 = (input[i] > (37+offset0));
int t0_0_9 = rt0_0_9 * (2 + (input[i] > (38+offset0)));
t0_0_9 += (1 - rt0_0_9) * (input[i] > (36+offset0));

int rt0_0_10 = (input[i] > (41+offset0));
int t0_0_10 = rt0_0_10 * (2 + (input[i] > (42+offset0)));
t0_0_10 += (1 - rt0_0_10) * (input[i] > (40+offset0));

int rt0_0_11 = (input[i] > (45+offset0));
int t0_0_11 = rt0_0_11 * (2 + (input[i] > (46+offset0)));
t0_0_11 += (1 - rt0_0_11) * (input[i] > (44+offset0));

int rt0_0_12 = (input[i] > (49+offset0));
int t0_0_12 = rt0_0_12 * (2 + (input[i] > (50+offset0)));
t0_0_12 += (1 - rt0_0_12) * (input[i] > (48+offset0));

int rt0_0_13 = (input[i] > (53+offset0));
int t0_0_13 = rt0_0_13 * (2 + (input[i] > (54+offset0)));
t0_0_13 += (1 - rt0_0_13) * (input[i] > (52+offset0));

int rt0_0_14 = (input[i] > (57+offset0));
int t0_0_14 = rt0_0_14 * (2 + (input[i] > (58+offset0)));
t0_0_14 += (1 - rt0_0_14) * (input[i] > (56+offset0));

int rt0_0_15 = (input[i] > (61+offset0));
int t0_0_15 = rt0_0_15 * (2 + (input[i] > (62+offset0)));
t0_0_15 += (1 - rt0_0_15) * (input[i] > (60+offset0));

int rt0_0_16 = (input[i] > (65+offset0));
int t0_0_16 = rt0_0_16 * (2 + (input[i] > (66+offset0)));
t0_0_16 += (1 - rt0_0_16) * (input[i] > (64+offset0));

int rt0_0_17 = (input[i] > (69+offset0));
int t0_0_17 = rt0_0_17 * (2 + (input[i] > (70+offset0)));
t0_0_17 += (1 - rt0_0_17) * (input[i] > (68+offset0));

int rt0_0_18 = (input[i] > (73+offset0));
int t0_0_18 = rt0_0_18 * (2 + (input[i] > (74+offset0)));
t0_0_18 += (1 - rt0_0_18) * (input[i] > (72+offset0));

int rt0_0_19 = (input[i] > (77+offset0));
int t0_0_19 = rt0_0_19 * (2 + (input[i] > (78+offset0)));
t0_0_19 += (1 - rt0_0_19) * (input[i] > (76+offset0));

int rt0_0_20 = (input[i] > (81+offset0));
int t0_0_20 = rt0_0_20 * (2 + (input[i] > (82+offset0)));
t0_0_20 += (1 - rt0_0_20) * (input[i] > (80+offset0));

int rt0_0_21 = (input[i] > (85+offset0));
int t0_0_21 = rt0_0_21 * (2 + (input[i] > (86+offset0)));
t0_0_21 += (1 - rt0_0_21) * (input[i] > (84+offset0));

int rt0_0_22 = (input[i] > (89+offset0));
int t0_0_22 = rt0_0_22 * (2 + (input[i] > (90+offset0)));
t0_0_22 += (1 - rt0_0_22) * (input[i] > (88+offset0));

int rt0_0_23 = (input[i] > (93+offset0));
int t0_0_23 = rt0_0_23 * (2 + (input[i] > (94+offset0)));
t0_0_23 += (1 - rt0_0_23) * (input[i] > (92+offset0));

int rt0_0_24 = (input[i] > (97+offset0));
int t0_0_24 = rt0_0_24 * (2 + (input[i] > (98+offset0)));
t0_0_24 += (1 - rt0_0_24) * (input[i] > (96+offset0));

int rt0_0_25 = (input[i] > (101+offset0));
int t0_0_25 = rt0_0_25 * (2 + (input[i] > (102+offset0)));
t0_0_25 += (1 - rt0_0_25) * (input[i] > (100+offset0));

int rt0_0_26 = (input[i] > (105+offset0));
int t0_0_26 = rt0_0_26 * (2 + (input[i] > (106+offset0)));
t0_0_26 += (1 - rt0_0_26) * (input[i] > (104+offset0));

int rt0_0_27 = (input[i] > (109+offset0));
int t0_0_27 = rt0_0_27 * (2 + (input[i] > (110+offset0)));
t0_0_27 += (1 - rt0_0_27) * (input[i] > (108+offset0));

int rt0_0_28 = (input[i] > (113+offset0));
int t0_0_28 = rt0_0_28 * (2 + (input[i] > (114+offset0)));
t0_0_28 += (1 - rt0_0_28) * (input[i] > (112+offset0));

int rt0_0_29 = (input[i] > (117+offset0));
int t0_0_29 = rt0_0_29 * (2 + (input[i] > (118+offset0)));
t0_0_29 += (1 - rt0_0_29) * (input[i] > (116+offset0));

int rt0_0_30 = (input[i] > (121+offset0));
int t0_0_30 = rt0_0_30 * (2 + (input[i] > (122+offset0)));
t0_0_30 += (1 - rt0_0_30) * (input[i] > (120+offset0));

int rt0_0_31 = (input[i] > (125+offset0));
int t0_0_31 = rt0_0_31 * (2 + (input[i] > (126+offset0)));
t0_0_31 += (1 - rt0_0_31) * (input[i] > (124+offset0));

int rt0_0_32 = (input[i] > (129+offset0));
int t0_0_32 = rt0_0_32 * (2 + (input[i] > (130+offset0)));
t0_0_32 += (1 - rt0_0_32) * (input[i] > (128+offset0));

int rt0_0_33 = (input[i] > (133+offset0));
int t0_0_33 = rt0_0_33 * (2 + (input[i] > (134+offset0)));
t0_0_33 += (1 - rt0_0_33) * (input[i] > (132+offset0));

int rt0_0_34 = (input[i] > (137+offset0));
int t0_0_34 = rt0_0_34 * (2 + (input[i] > (138+offset0)));
t0_0_34 += (1 - rt0_0_34) * (input[i] > (136+offset0));

int rt0_0_35 = (input[i] > (141+offset0));
int t0_0_35 = rt0_0_35 * (2 + (input[i] > (142+offset0)));
t0_0_35 += (1 - rt0_0_35) * (input[i] > (140+offset0));

int rt0_0_36 = (input[i] > (145+offset0));
int t0_0_36 = rt0_0_36 * (2 + (input[i] > (146+offset0)));
t0_0_36 += (1 - rt0_0_36) * (input[i] > (144+offset0));

int rt0_0_37 = (input[i] > (149+offset0));
int t0_0_37 = rt0_0_37 * (2 + (input[i] > (150+offset0)));
t0_0_37 += (1 - rt0_0_37) * (input[i] > (148+offset0));

int rt0_0_38 = (input[i] > (153+offset0));
int t0_0_38 = rt0_0_38 * (2 + (input[i] > (154+offset0)));
t0_0_38 += (1 - rt0_0_38) * (input[i] > (152+offset0));

int rt0_0_39 = (input[i] > (157+offset0));
int t0_0_39 = rt0_0_39 * (2 + (input[i] > (158+offset0)));
t0_0_39 += (1 - rt0_0_39) * (input[i] > (156+offset0));

int rt0_0_40 = (input[i] > (161+offset0));
int t0_0_40 = rt0_0_40 * (2 + (input[i] > (162+offset0)));
t0_0_40 += (1 - rt0_0_40) * (input[i] > (160+offset0));

int rt0_0_41 = (input[i] > (165+offset0));
int t0_0_41 = rt0_0_41 * (2 + (input[i] > (166+offset0)));
t0_0_41 += (1 - rt0_0_41) * (input[i] > (164+offset0));

int rt0_0_42 = (input[i] > (169+offset0));
int t0_0_42 = rt0_0_42 * (2 + (input[i] > (170+offset0)));
t0_0_42 += (1 - rt0_0_42) * (input[i] > (168+offset0));

int rt0_0_43 = (input[i] > (173+offset0));
int t0_0_43 = rt0_0_43 * (2 + (input[i] > (174+offset0)));
t0_0_43 += (1 - rt0_0_43) * (input[i] > (172+offset0));

int rt0_0_44 = (input[i] > (177+offset0));
int t0_0_44 = rt0_0_44 * (2 + (input[i] > (178+offset0)));
t0_0_44 += (1 - rt0_0_44) * (input[i] > (176+offset0));

int rt0_0_45 = (input[i] > (181+offset0));
int t0_0_45 = rt0_0_45 * (2 + (input[i] > (182+offset0)));
t0_0_45 += (1 - rt0_0_45) * (input[i] > (180+offset0));

int rt0_0_46 = (input[i] > (185+offset0));
int t0_0_46 = rt0_0_46 * (2 + (input[i] > (186+offset0)));
t0_0_46 += (1 - rt0_0_46) * (input[i] > (184+offset0));

int rt0_0_47 = (input[i] > (189+offset0));
int t0_0_47 = rt0_0_47 * (2 + (input[i] > (190+offset0)));
t0_0_47 += (1 - rt0_0_47) * (input[i] > (188+offset0));

int rt0_0_48 = (input[i] > (193+offset0));
int t0_0_48 = rt0_0_48 * (2 + (input[i] > (194+offset0)));
t0_0_48 += (1 - rt0_0_48) * (input[i] > (192+offset0));

int rt0_0_49 = (input[i] > (197+offset0));
int t0_0_49 = rt0_0_49 * (2 + (input[i] > (198+offset0)));
t0_0_49 += (1 - rt0_0_49) * (input[i] > (196+offset0));

int rt0_0_50 = (input[i] > (201+offset0));
int t0_0_50 = rt0_0_50 * (2 + (input[i] > (202+offset0)));
t0_0_50 += (1 - rt0_0_50) * (input[i] > (200+offset0));

int rt0_0_51 = (input[i] > (205+offset0));
int t0_0_51 = rt0_0_51 * (2 + (input[i] > (206+offset0)));
t0_0_51 += (1 - rt0_0_51) * (input[i] > (204+offset0));

int rt0_0_52 = (input[i] > (209+offset0));
int t0_0_52 = rt0_0_52 * (2 + (input[i] > (210+offset0)));
t0_0_52 += (1 - rt0_0_52) * (input[i] > (208+offset0));

int rt0_0_53 = (input[i] > (213+offset0));
int t0_0_53 = rt0_0_53 * (2 + (input[i] > (214+offset0)));
t0_0_53 += (1 - rt0_0_53) * (input[i] > (212+offset0));

int rt0_0_54 = (input[i] > (217+offset0));
int t0_0_54 = rt0_0_54 * (2 + (input[i] > (218+offset0)));
t0_0_54 += (1 - rt0_0_54) * (input[i] > (216+offset0));

int rt0_0_55 = (input[i] > (221+offset0));
int t0_0_55 = rt0_0_55 * (2 + (input[i] > (222+offset0)));
t0_0_55 += (1 - rt0_0_55) * (input[i] > (220+offset0));

int rt0_0_56 = (input[i] > (225+offset0));
int t0_0_56 = rt0_0_56 * (2 + (input[i] > (226+offset0)));
t0_0_56 += (1 - rt0_0_56) * (input[i] > (224+offset0));

int rt0_0_57 = (input[i] > (229+offset0));
int t0_0_57 = rt0_0_57 * (2 + (input[i] > (230+offset0)));
t0_0_57 += (1 - rt0_0_57) * (input[i] > (228+offset0));

int rt0_0_58 = (input[i] > (233+offset0));
int t0_0_58 = rt0_0_58 * (2 + (input[i] > (234+offset0)));
t0_0_58 += (1 - rt0_0_58) * (input[i] > (232+offset0));

int rt0_0_59 = (input[i] > (237+offset0));
int t0_0_59 = rt0_0_59 * (2 + (input[i] > (238+offset0)));
t0_0_59 += (1 - rt0_0_59) * (input[i] > (236+offset0));

int rt0_0_60 = (input[i] > (241+offset0));
int t0_0_60 = rt0_0_60 * (2 + (input[i] > (242+offset0)));
t0_0_60 += (1 - rt0_0_60) * (input[i] > (240+offset0));

int rt0_0_61 = (input[i] > (245+offset0));
int t0_0_61 = rt0_0_61 * (2 + (input[i] > (246+offset0)));
t0_0_61 += (1 - rt0_0_61) * (input[i] > (244+offset0));

int rt0_0_62 = (input[i] > (249+offset0));
int t0_0_62 = rt0_0_62 * (2 + (input[i] > (250+offset0)));
t0_0_62 += (1 - rt0_0_62) * (input[i] > (248+offset0));

int rt0_0_63 = (input[i] > (253+offset0));
int t0_0_63 = rt0_0_63 * (2 + (input[i] > (254+offset0)));
t0_0_63 += (1 - rt0_0_63) * (input[i] > (252+offset0));

int rt0_0_64 = (input[i] > (257+offset0));
int t0_0_64 = rt0_0_64 * (2 + (input[i] > (258+offset0)));
t0_0_64 += (1 - rt0_0_64) * (input[i] > (256+offset0));

int rt0_0_65 = (input[i] > (261+offset0));
int t0_0_65 = rt0_0_65 * (2 + (input[i] > (262+offset0)));
t0_0_65 += (1 - rt0_0_65) * (input[i] > (260+offset0));

int rt0_0_66 = (input[i] > (265+offset0));
int t0_0_66 = rt0_0_66 * (2 + (input[i] > (266+offset0)));
t0_0_66 += (1 - rt0_0_66) * (input[i] > (264+offset0));

int rt0_0_67 = (input[i] > (269+offset0));
int t0_0_67 = rt0_0_67 * (2 + (input[i] > (270+offset0)));
t0_0_67 += (1 - rt0_0_67) * (input[i] > (268+offset0));

int rt0_0_68 = (input[i] > (273+offset0));
int t0_0_68 = rt0_0_68 * (2 + (input[i] > (274+offset0)));
t0_0_68 += (1 - rt0_0_68) * (input[i] > (272+offset0));

int rt0_0_69 = (input[i] > (277+offset0));
int t0_0_69 = rt0_0_69 * (2 + (input[i] > (278+offset0)));
t0_0_69 += (1 - rt0_0_69) * (input[i] > (276+offset0));

int rt0_0_70 = (input[i] > (281+offset0));
int t0_0_70 = rt0_0_70 * (2 + (input[i] > (282+offset0)));
t0_0_70 += (1 - rt0_0_70) * (input[i] > (280+offset0));

int rt0_0_71 = (input[i] > (285+offset0));
int t0_0_71 = rt0_0_71 * (2 + (input[i] > (286+offset0)));
t0_0_71 += (1 - rt0_0_71) * (input[i] > (284+offset0));

int rt0_0_72 = (input[i] > (289+offset0));
int t0_0_72 = rt0_0_72 * (2 + (input[i] > (290+offset0)));
t0_0_72 += (1 - rt0_0_72) * (input[i] > (288+offset0));

int rt0_0_73 = (input[i] > (293+offset0));
int t0_0_73 = rt0_0_73 * (2 + (input[i] > (294+offset0)));
t0_0_73 += (1 - rt0_0_73) * (input[i] > (292+offset0));

int rt0_0_74 = (input[i] > (297+offset0));
int t0_0_74 = rt0_0_74 * (2 + (input[i] > (298+offset0)));
t0_0_74 += (1 - rt0_0_74) * (input[i] > (296+offset0));

int rt0_0_75 = (input[i] > (301+offset0));
int t0_0_75 = rt0_0_75 * (2 + (input[i] > (302+offset0)));
t0_0_75 += (1 - rt0_0_75) * (input[i] > (300+offset0));

int rt0_0_76 = (input[i] > (305+offset0));
int t0_0_76 = rt0_0_76 * (2 + (input[i] > (306+offset0)));
t0_0_76 += (1 - rt0_0_76) * (input[i] > (304+offset0));

int rt0_0_77 = (input[i] > (309+offset0));
int t0_0_77 = rt0_0_77 * (2 + (input[i] > (310+offset0)));
t0_0_77 += (1 - rt0_0_77) * (input[i] > (308+offset0));

int rt0_0_78 = (input[i] > (313+offset0));
int t0_0_78 = rt0_0_78 * (2 + (input[i] > (314+offset0)));
t0_0_78 += (1 - rt0_0_78) * (input[i] > (312+offset0));

int rt0_0_79 = (input[i] > (317+offset0));
int t0_0_79 = rt0_0_79 * (2 + (input[i] > (318+offset0)));
t0_0_79 += (1 - rt0_0_79) * (input[i] > (316+offset0));

int rt0_0_80 = (input[i] > (321+offset0));
int t0_0_80 = rt0_0_80 * (2 + (input[i] > (322+offset0)));
t0_0_80 += (1 - rt0_0_80) * (input[i] > (320+offset0));

int rt0_0_81 = (input[i] > (325+offset0));
int t0_0_81 = rt0_0_81 * (2 + (input[i] > (326+offset0)));
t0_0_81 += (1 - rt0_0_81) * (input[i] > (324+offset0));

int rt0_0_82 = (input[i] > (329+offset0));
int t0_0_82 = rt0_0_82 * (2 + (input[i] > (330+offset0)));
t0_0_82 += (1 - rt0_0_82) * (input[i] > (328+offset0));

int rt0_0_83 = (input[i] > (333+offset0));
int t0_0_83 = rt0_0_83 * (2 + (input[i] > (334+offset0)));
t0_0_83 += (1 - rt0_0_83) * (input[i] > (332+offset0));

int rt0_0_84 = (input[i] > (337+offset0));
int t0_0_84 = rt0_0_84 * (2 + (input[i] > (338+offset0)));
t0_0_84 += (1 - rt0_0_84) * (input[i] > (336+offset0));

int rt0_0_85 = (input[i] > (341+offset0));
int t0_0_85 = rt0_0_85 * (2 + (input[i] > (342+offset0)));
t0_0_85 += (1 - rt0_0_85) * (input[i] > (340+offset0));

int rt0_0_86 = (input[i] > (345+offset0));
int t0_0_86 = rt0_0_86 * (2 + (input[i] > (346+offset0)));
t0_0_86 += (1 - rt0_0_86) * (input[i] > (344+offset0));

int rt0_0_87 = (input[i] > (349+offset0));
int t0_0_87 = rt0_0_87 * (2 + (input[i] > (350+offset0)));
t0_0_87 += (1 - rt0_0_87) * (input[i] > (348+offset0));

int rt0_0_88 = (input[i] > (353+offset0));
int t0_0_88 = rt0_0_88 * (2 + (input[i] > (354+offset0)));
t0_0_88 += (1 - rt0_0_88) * (input[i] > (352+offset0));

int rt0_0_89 = (input[i] > (357+offset0));
int t0_0_89 = rt0_0_89 * (2 + (input[i] > (358+offset0)));
t0_0_89 += (1 - rt0_0_89) * (input[i] > (356+offset0));

int rt0_0_90 = (input[i] > (361+offset0));
int t0_0_90 = rt0_0_90 * (2 + (input[i] > (362+offset0)));
t0_0_90 += (1 - rt0_0_90) * (input[i] > (360+offset0));

int rt0_0_91 = (input[i] > (365+offset0));
int t0_0_91 = rt0_0_91 * (2 + (input[i] > (366+offset0)));
t0_0_91 += (1 - rt0_0_91) * (input[i] > (364+offset0));

int rt0_0_92 = (input[i] > (369+offset0));
int t0_0_92 = rt0_0_92 * (2 + (input[i] > (370+offset0)));
t0_0_92 += (1 - rt0_0_92) * (input[i] > (368+offset0));

int rt0_0_93 = (input[i] > (373+offset0));
int t0_0_93 = rt0_0_93 * (2 + (input[i] > (374+offset0)));
t0_0_93 += (1 - rt0_0_93) * (input[i] > (372+offset0));

int rt0_0_94 = (input[i] > (377+offset0));
int t0_0_94 = rt0_0_94 * (2 + (input[i] > (378+offset0)));
t0_0_94 += (1 - rt0_0_94) * (input[i] > (376+offset0));

int rt0_0_95 = (input[i] > (381+offset0));
int t0_0_95 = rt0_0_95 * (2 + (input[i] > (382+offset0)));
t0_0_95 += (1 - rt0_0_95) * (input[i] > (380+offset0));

int rt0_0_96 = (input[i] > (385+offset0));
int t0_0_96 = rt0_0_96 * (2 + (input[i] > (386+offset0)));
t0_0_96 += (1 - rt0_0_96) * (input[i] > (384+offset0));

int rt0_0_97 = (input[i] > (389+offset0));
int t0_0_97 = rt0_0_97 * (2 + (input[i] > (390+offset0)));
t0_0_97 += (1 - rt0_0_97) * (input[i] > (388+offset0));

int rt0_0_98 = (input[i] > (393+offset0));
int t0_0_98 = rt0_0_98 * (2 + (input[i] > (394+offset0)));
t0_0_98 += (1 - rt0_0_98) * (input[i] > (392+offset0));

int rt0_0_99 = (input[i] > (397+offset0));
int t0_0_99 = rt0_0_99 * (2 + (input[i] > (398+offset0)));
t0_0_99 += (1 - rt0_0_99) * (input[i] > (396+offset0));

int rt0_0_100 = (input[i] > (401+offset0));
int t0_0_100 = rt0_0_100 * (2 + (input[i] > (402+offset0)));
t0_0_100 += (1 - rt0_0_100) * (input[i] > (400+offset0));

int rt0_0_101 = (input[i] > (405+offset0));
int t0_0_101 = rt0_0_101 * (2 + (input[i] > (406+offset0)));
t0_0_101 += (1 - rt0_0_101) * (input[i] > (404+offset0));

int rt0_0_102 = (input[i] > (409+offset0));
int t0_0_102 = rt0_0_102 * (2 + (input[i] > (410+offset0)));
t0_0_102 += (1 - rt0_0_102) * (input[i] > (408+offset0));

int rt0_0_103 = (input[i] > (413+offset0));
int t0_0_103 = rt0_0_103 * (2 + (input[i] > (414+offset0)));
t0_0_103 += (1 - rt0_0_103) * (input[i] > (412+offset0));

int rt0_0_104 = (input[i] > (417+offset0));
int t0_0_104 = rt0_0_104 * (2 + (input[i] > (418+offset0)));
t0_0_104 += (1 - rt0_0_104) * (input[i] > (416+offset0));

int rt0_0_105 = (input[i] > (421+offset0));
int t0_0_105 = rt0_0_105 * (2 + (input[i] > (422+offset0)));
t0_0_105 += (1 - rt0_0_105) * (input[i] > (420+offset0));

int rt0_0_106 = (input[i] > (425+offset0));
int t0_0_106 = rt0_0_106 * (2 + (input[i] > (426+offset0)));
t0_0_106 += (1 - rt0_0_106) * (input[i] > (424+offset0));

int rt0_0_107 = (input[i] > (429+offset0));
int t0_0_107 = rt0_0_107 * (2 + (input[i] > (430+offset0)));
t0_0_107 += (1 - rt0_0_107) * (input[i] > (428+offset0));

int rt0_0_108 = (input[i] > (433+offset0));
int t0_0_108 = rt0_0_108 * (2 + (input[i] > (434+offset0)));
t0_0_108 += (1 - rt0_0_108) * (input[i] > (432+offset0));

int rt0_0_109 = (input[i] > (437+offset0));
int t0_0_109 = rt0_0_109 * (2 + (input[i] > (438+offset0)));
t0_0_109 += (1 - rt0_0_109) * (input[i] > (436+offset0));

int rt0_0_110 = (input[i] > (441+offset0));
int t0_0_110 = rt0_0_110 * (2 + (input[i] > (442+offset0)));
t0_0_110 += (1 - rt0_0_110) * (input[i] > (440+offset0));

int rt0_0_111 = (input[i] > (445+offset0));
int t0_0_111 = rt0_0_111 * (2 + (input[i] > (446+offset0)));
t0_0_111 += (1 - rt0_0_111) * (input[i] > (444+offset0));

int rt0_0_112 = (input[i] > (449+offset0));
int t0_0_112 = rt0_0_112 * (2 + (input[i] > (450+offset0)));
t0_0_112 += (1 - rt0_0_112) * (input[i] > (448+offset0));

int rt0_0_113 = (input[i] > (453+offset0));
int t0_0_113 = rt0_0_113 * (2 + (input[i] > (454+offset0)));
t0_0_113 += (1 - rt0_0_113) * (input[i] > (452+offset0));

int rt0_0_114 = (input[i] > (457+offset0));
int t0_0_114 = rt0_0_114 * (2 + (input[i] > (458+offset0)));
t0_0_114 += (1 - rt0_0_114) * (input[i] > (456+offset0));

int rt0_0_115 = (input[i] > (461+offset0));
int t0_0_115 = rt0_0_115 * (2 + (input[i] > (462+offset0)));
t0_0_115 += (1 - rt0_0_115) * (input[i] > (460+offset0));

int rt0_0_116 = (input[i] > (465+offset0));
int t0_0_116 = rt0_0_116 * (2 + (input[i] > (466+offset0)));
t0_0_116 += (1 - rt0_0_116) * (input[i] > (464+offset0));

int rt0_0_117 = (input[i] > (469+offset0));
int t0_0_117 = rt0_0_117 * (2 + (input[i] > (470+offset0)));
t0_0_117 += (1 - rt0_0_117) * (input[i] > (468+offset0));

int rt0_0_118 = (input[i] > (473+offset0));
int t0_0_118 = rt0_0_118 * (2 + (input[i] > (474+offset0)));
t0_0_118 += (1 - rt0_0_118) * (input[i] > (472+offset0));

int rt0_0_119 = (input[i] > (477+offset0));
int t0_0_119 = rt0_0_119 * (2 + (input[i] > (478+offset0)));
t0_0_119 += (1 - rt0_0_119) * (input[i] > (476+offset0));

int rt0_0_120 = (input[i] > (481+offset0));
int t0_0_120 = rt0_0_120 * (2 + (input[i] > (482+offset0)));
t0_0_120 += (1 - rt0_0_120) * (input[i] > (480+offset0));

int rt0_0_121 = (input[i] > (485+offset0));
int t0_0_121 = rt0_0_121 * (2 + (input[i] > (486+offset0)));
t0_0_121 += (1 - rt0_0_121) * (input[i] > (484+offset0));

int rt0_0_122 = (input[i] > (489+offset0));
int t0_0_122 = rt0_0_122 * (2 + (input[i] > (490+offset0)));
t0_0_122 += (1 - rt0_0_122) * (input[i] > (488+offset0));

int rt0_0_123 = (input[i] > (493+offset0));
int t0_0_123 = rt0_0_123 * (2 + (input[i] > (494+offset0)));
t0_0_123 += (1 - rt0_0_123) * (input[i] > (492+offset0));

int rt0_0_124 = (input[i] > (497+offset0));
int t0_0_124 = rt0_0_124 * (2 + (input[i] > (498+offset0)));
t0_0_124 += (1 - rt0_0_124) * (input[i] > (496+offset0));

int rt0_0_125 = (input[i] > (501+offset0));
int t0_0_125 = rt0_0_125 * (2 + (input[i] > (502+offset0)));
t0_0_125 += (1 - rt0_0_125) * (input[i] > (500+offset0));

int rt0_0_126 = (input[i] > (505+offset0));
int t0_0_126 = rt0_0_126 * (2 + (input[i] > (506+offset0)));
t0_0_126 += (1 - rt0_0_126) * (input[i] > (504+offset0));

int rt0_0_127 = (input[i] > (509+offset0));
int t0_0_127 = rt0_0_127 * (2 + (input[i] > (510+offset0)));
t0_0_127 += (1 - rt0_0_127) * (input[i] > (508+offset0));

int rt0_0_128 = (input[i] > (513+offset0));
int t0_0_128 = rt0_0_128 * (2 + (input[i] > (514+offset0)));
t0_0_128 += (1 - rt0_0_128) * (input[i] > (512+offset0));

int rt0_0_129 = (input[i] > (517+offset0));
int t0_0_129 = rt0_0_129 * (2 + (input[i] > (518+offset0)));
t0_0_129 += (1 - rt0_0_129) * (input[i] > (516+offset0));

int rt0_0_130 = (input[i] > (521+offset0));
int t0_0_130 = rt0_0_130 * (2 + (input[i] > (522+offset0)));
t0_0_130 += (1 - rt0_0_130) * (input[i] > (520+offset0));

int rt0_0_131 = (input[i] > (525+offset0));
int t0_0_131 = rt0_0_131 * (2 + (input[i] > (526+offset0)));
t0_0_131 += (1 - rt0_0_131) * (input[i] > (524+offset0));

int rt0_0_132 = (input[i] > (529+offset0));
int t0_0_132 = rt0_0_132 * (2 + (input[i] > (530+offset0)));
t0_0_132 += (1 - rt0_0_132) * (input[i] > (528+offset0));

int rt0_0_133 = (input[i] > (533+offset0));
int t0_0_133 = rt0_0_133 * (2 + (input[i] > (534+offset0)));
t0_0_133 += (1 - rt0_0_133) * (input[i] > (532+offset0));

int rt0_0_134 = (input[i] > (537+offset0));
int t0_0_134 = rt0_0_134 * (2 + (input[i] > (538+offset0)));
t0_0_134 += (1 - rt0_0_134) * (input[i] > (536+offset0));

int rt0_0_135 = (input[i] > (541+offset0));
int t0_0_135 = rt0_0_135 * (2 + (input[i] > (542+offset0)));
t0_0_135 += (1 - rt0_0_135) * (input[i] > (540+offset0));

int rt0_0_136 = (input[i] > (545+offset0));
int t0_0_136 = rt0_0_136 * (2 + (input[i] > (546+offset0)));
t0_0_136 += (1 - rt0_0_136) * (input[i] > (544+offset0));

int rt0_0_137 = (input[i] > (549+offset0));
int t0_0_137 = rt0_0_137 * (2 + (input[i] > (550+offset0)));
t0_0_137 += (1 - rt0_0_137) * (input[i] > (548+offset0));

int rt0_0_138 = (input[i] > (553+offset0));
int t0_0_138 = rt0_0_138 * (2 + (input[i] > (554+offset0)));
t0_0_138 += (1 - rt0_0_138) * (input[i] > (552+offset0));

int rt0_0_139 = (input[i] > (557+offset0));
int t0_0_139 = rt0_0_139 * (2 + (input[i] > (558+offset0)));
t0_0_139 += (1 - rt0_0_139) * (input[i] > (556+offset0));

int rt0_0_140 = (input[i] > (561+offset0));
int t0_0_140 = rt0_0_140 * (2 + (input[i] > (562+offset0)));
t0_0_140 += (1 - rt0_0_140) * (input[i] > (560+offset0));

int rt0_0_141 = (input[i] > (565+offset0));
int t0_0_141 = rt0_0_141 * (2 + (input[i] > (566+offset0)));
t0_0_141 += (1 - rt0_0_141) * (input[i] > (564+offset0));

int rt0_0_142 = (input[i] > (569+offset0));
int t0_0_142 = rt0_0_142 * (2 + (input[i] > (570+offset0)));
t0_0_142 += (1 - rt0_0_142) * (input[i] > (568+offset0));

int rt0_0_143 = (input[i] > (573+offset0));
int t0_0_143 = rt0_0_143 * (2 + (input[i] > (574+offset0)));
t0_0_143 += (1 - rt0_0_143) * (input[i] > (572+offset0));

int rt0_0_144 = (input[i] > (577+offset0));
int t0_0_144 = rt0_0_144 * (2 + (input[i] > (578+offset0)));
t0_0_144 += (1 - rt0_0_144) * (input[i] > (576+offset0));

int rt0_0_145 = (input[i] > (581+offset0));
int t0_0_145 = rt0_0_145 * (2 + (input[i] > (582+offset0)));
t0_0_145 += (1 - rt0_0_145) * (input[i] > (580+offset0));

int rt0_0_146 = (input[i] > (585+offset0));
int t0_0_146 = rt0_0_146 * (2 + (input[i] > (586+offset0)));
t0_0_146 += (1 - rt0_0_146) * (input[i] > (584+offset0));

int rt0_0_147 = (input[i] > (589+offset0));
int t0_0_147 = rt0_0_147 * (2 + (input[i] > (590+offset0)));
t0_0_147 += (1 - rt0_0_147) * (input[i] > (588+offset0));

int rt0_0_148 = (input[i] > (593+offset0));
int t0_0_148 = rt0_0_148 * (2 + (input[i] > (594+offset0)));
t0_0_148 += (1 - rt0_0_148) * (input[i] > (592+offset0));

int rt0_0_149 = (input[i] > (597+offset0));
int t0_0_149 = rt0_0_149 * (2 + (input[i] > (598+offset0)));
t0_0_149 += (1 - rt0_0_149) * (input[i] > (596+offset0));

int rt0_0_150 = (input[i] > (601+offset0));
int t0_0_150 = rt0_0_150 * (2 + (input[i] > (602+offset0)));
t0_0_150 += (1 - rt0_0_150) * (input[i] > (600+offset0));

int rt0_0_151 = (input[i] > (605+offset0));
int t0_0_151 = rt0_0_151 * (2 + (input[i] > (606+offset0)));
t0_0_151 += (1 - rt0_0_151) * (input[i] > (604+offset0));

int rt0_0_152 = (input[i] > (609+offset0));
int t0_0_152 = rt0_0_152 * (2 + (input[i] > (610+offset0)));
t0_0_152 += (1 - rt0_0_152) * (input[i] > (608+offset0));

int rt0_0_153 = (input[i] > (613+offset0));
int t0_0_153 = rt0_0_153 * (2 + (input[i] > (614+offset0)));
t0_0_153 += (1 - rt0_0_153) * (input[i] > (612+offset0));

int rt0_0_154 = (input[i] > (617+offset0));
int t0_0_154 = rt0_0_154 * (2 + (input[i] > (618+offset0)));
t0_0_154 += (1 - rt0_0_154) * (input[i] > (616+offset0));

int rt0_0_155 = (input[i] > (621+offset0));
int t0_0_155 = rt0_0_155 * (2 + (input[i] > (622+offset0)));
t0_0_155 += (1 - rt0_0_155) * (input[i] > (620+offset0));

int rt0_0_156 = (input[i] > (625+offset0));
int t0_0_156 = rt0_0_156 * (2 + (input[i] > (626+offset0)));
t0_0_156 += (1 - rt0_0_156) * (input[i] > (624+offset0));

int rt0_0_157 = (input[i] > (629+offset0));
int t0_0_157 = rt0_0_157 * (2 + (input[i] > (630+offset0)));
t0_0_157 += (1 - rt0_0_157) * (input[i] > (628+offset0));

int rt0_0_158 = (input[i] > (633+offset0));
int t0_0_158 = rt0_0_158 * (2 + (input[i] > (634+offset0)));
t0_0_158 += (1 - rt0_0_158) * (input[i] > (632+offset0));

int rt0_0_159 = (input[i] > (637+offset0));
int t0_0_159 = rt0_0_159 * (2 + (input[i] > (638+offset0)));
t0_0_159 += (1 - rt0_0_159) * (input[i] > (636+offset0));

int rt0_0_160 = (input[i] > (641+offset0));
int t0_0_160 = rt0_0_160 * (2 + (input[i] > (642+offset0)));
t0_0_160 += (1 - rt0_0_160) * (input[i] > (640+offset0));

int rt0_0_161 = (input[i] > (645+offset0));
int t0_0_161 = rt0_0_161 * (2 + (input[i] > (646+offset0)));
t0_0_161 += (1 - rt0_0_161) * (input[i] > (644+offset0));

int rt0_0_162 = (input[i] > (649+offset0));
int t0_0_162 = rt0_0_162 * (2 + (input[i] > (650+offset0)));
t0_0_162 += (1 - rt0_0_162) * (input[i] > (648+offset0));

int rt0_0_163 = (input[i] > (653+offset0));
int t0_0_163 = rt0_0_163 * (2 + (input[i] > (654+offset0)));
t0_0_163 += (1 - rt0_0_163) * (input[i] > (652+offset0));

int rt0_0_164 = (input[i] > (657+offset0));
int t0_0_164 = rt0_0_164 * (2 + (input[i] > (658+offset0)));
t0_0_164 += (1 - rt0_0_164) * (input[i] > (656+offset0));

int rt0_0_165 = (input[i] > (661+offset0));
int t0_0_165 = rt0_0_165 * (2 + (input[i] > (662+offset0)));
t0_0_165 += (1 - rt0_0_165) * (input[i] > (660+offset0));

int rt0_0_166 = (input[i] > (665+offset0));
int t0_0_166 = rt0_0_166 * (2 + (input[i] > (666+offset0)));
t0_0_166 += (1 - rt0_0_166) * (input[i] > (664+offset0));

int rt0_0_167 = (input[i] > (669+offset0));
int t0_0_167 = rt0_0_167 * (2 + (input[i] > (670+offset0)));
t0_0_167 += (1 - rt0_0_167) * (input[i] > (668+offset0));

int rt0_0_168 = (input[i] > (673+offset0));
int t0_0_168 = rt0_0_168 * (2 + (input[i] > (674+offset0)));
t0_0_168 += (1 - rt0_0_168) * (input[i] > (672+offset0));

int rt0_0_169 = (input[i] > (677+offset0));
int t0_0_169 = rt0_0_169 * (2 + (input[i] > (678+offset0)));
t0_0_169 += (1 - rt0_0_169) * (input[i] > (676+offset0));

int rt0_0_170 = (input[i] > (681+offset0));
int t0_0_170 = rt0_0_170 * (2 + (input[i] > (682+offset0)));
t0_0_170 += (1 - rt0_0_170) * (input[i] > (680+offset0));

int rt0_0_171 = (input[i] > (685+offset0));
int t0_0_171 = rt0_0_171 * (2 + (input[i] > (686+offset0)));
t0_0_171 += (1 - rt0_0_171) * (input[i] > (684+offset0));

int rt0_0_172 = (input[i] > (689+offset0));
int t0_0_172 = rt0_0_172 * (2 + (input[i] > (690+offset0)));
t0_0_172 += (1 - rt0_0_172) * (input[i] > (688+offset0));

int rt0_0_173 = (input[i] > (693+offset0));
int t0_0_173 = rt0_0_173 * (2 + (input[i] > (694+offset0)));
t0_0_173 += (1 - rt0_0_173) * (input[i] > (692+offset0));

int rt0_0_174 = (input[i] > (697+offset0));
int t0_0_174 = rt0_0_174 * (2 + (input[i] > (698+offset0)));
t0_0_174 += (1 - rt0_0_174) * (input[i] > (696+offset0));

int rt0_0_175 = (input[i] > (701+offset0));
int t0_0_175 = rt0_0_175 * (2 + (input[i] > (702+offset0)));
t0_0_175 += (1 - rt0_0_175) * (input[i] > (700+offset0));

int rt0_0_176 = (input[i] > (705+offset0));
int t0_0_176 = rt0_0_176 * (2 + (input[i] > (706+offset0)));
t0_0_176 += (1 - rt0_0_176) * (input[i] > (704+offset0));

int rt0_0_177 = (input[i] > (709+offset0));
int t0_0_177 = rt0_0_177 * (2 + (input[i] > (710+offset0)));
t0_0_177 += (1 - rt0_0_177) * (input[i] > (708+offset0));

int rt0_0_178 = (input[i] > (713+offset0));
int t0_0_178 = rt0_0_178 * (2 + (input[i] > (714+offset0)));
t0_0_178 += (1 - rt0_0_178) * (input[i] > (712+offset0));

int rt0_0_179 = (input[i] > (717+offset0));
int t0_0_179 = rt0_0_179 * (2 + (input[i] > (718+offset0)));
t0_0_179 += (1 - rt0_0_179) * (input[i] > (716+offset0));

int rt0_0_180 = (input[i] > (721+offset0));
int t0_0_180 = rt0_0_180 * (2 + (input[i] > (722+offset0)));
t0_0_180 += (1 - rt0_0_180) * (input[i] > (720+offset0));

int rt0_0_181 = (input[i] > (725+offset0));
int t0_0_181 = rt0_0_181 * (2 + (input[i] > (726+offset0)));
t0_0_181 += (1 - rt0_0_181) * (input[i] > (724+offset0));

int rt0_0_182 = (input[i] > (729+offset0));
int t0_0_182 = rt0_0_182 * (2 + (input[i] > (730+offset0)));
t0_0_182 += (1 - rt0_0_182) * (input[i] > (728+offset0));

int rt0_0_183 = (input[i] > (733+offset0));
int t0_0_183 = rt0_0_183 * (2 + (input[i] > (734+offset0)));
t0_0_183 += (1 - rt0_0_183) * (input[i] > (732+offset0));

int rt0_0_184 = (input[i] > (737+offset0));
int t0_0_184 = rt0_0_184 * (2 + (input[i] > (738+offset0)));
t0_0_184 += (1 - rt0_0_184) * (input[i] > (736+offset0));

int rt0_0_185 = (input[i] > (741+offset0));
int t0_0_185 = rt0_0_185 * (2 + (input[i] > (742+offset0)));
t0_0_185 += (1 - rt0_0_185) * (input[i] > (740+offset0));

int rt0_0_186 = (input[i] > (745+offset0));
int t0_0_186 = rt0_0_186 * (2 + (input[i] > (746+offset0)));
t0_0_186 += (1 - rt0_0_186) * (input[i] > (744+offset0));

int rt0_0_187 = (input[i] > (749+offset0));
int t0_0_187 = rt0_0_187 * (2 + (input[i] > (750+offset0)));
t0_0_187 += (1 - rt0_0_187) * (input[i] > (748+offset0));

int rt0_0_188 = (input[i] > (753+offset0));
int t0_0_188 = rt0_0_188 * (2 + (input[i] > (754+offset0)));
t0_0_188 += (1 - rt0_0_188) * (input[i] > (752+offset0));

int rt0_0_189 = (input[i] > (757+offset0));
int t0_0_189 = rt0_0_189 * (2 + (input[i] > (758+offset0)));
t0_0_189 += (1 - rt0_0_189) * (input[i] > (756+offset0));

int rt0_0_190 = (input[i] > (761+offset0));
int t0_0_190 = rt0_0_190 * (2 + (input[i] > (762+offset0)));
t0_0_190 += (1 - rt0_0_190) * (input[i] > (760+offset0));

int rt0_0_191 = (input[i] > (765+offset0));
int t0_0_191 = rt0_0_191 * (2 + (input[i] > (766+offset0)));
t0_0_191 += (1 - rt0_0_191) * (input[i] > (764+offset0));

int rt0_0_192 = (input[i] > (769+offset0));
int t0_0_192 = rt0_0_192 * (2 + (input[i] > (770+offset0)));
t0_0_192 += (1 - rt0_0_192) * (input[i] > (768+offset0));

int rt0_0_193 = (input[i] > (773+offset0));
int t0_0_193 = rt0_0_193 * (2 + (input[i] > (774+offset0)));
t0_0_193 += (1 - rt0_0_193) * (input[i] > (772+offset0));

int rt0_0_194 = (input[i] > (777+offset0));
int t0_0_194 = rt0_0_194 * (2 + (input[i] > (778+offset0)));
t0_0_194 += (1 - rt0_0_194) * (input[i] > (776+offset0));

int rt0_0_195 = (input[i] > (781+offset0));
int t0_0_195 = rt0_0_195 * (2 + (input[i] > (782+offset0)));
t0_0_195 += (1 - rt0_0_195) * (input[i] > (780+offset0));

int rt0_0_196 = (input[i] > (785+offset0));
int t0_0_196 = rt0_0_196 * (2 + (input[i] > (786+offset0)));
t0_0_196 += (1 - rt0_0_196) * (input[i] > (784+offset0));

int rt0_0_197 = (input[i] > (789+offset0));
int t0_0_197 = rt0_0_197 * (2 + (input[i] > (790+offset0)));
t0_0_197 += (1 - rt0_0_197) * (input[i] > (788+offset0));

int rt0_0_198 = (input[i] > (793+offset0));
int t0_0_198 = rt0_0_198 * (2 + (input[i] > (794+offset0)));
t0_0_198 += (1 - rt0_0_198) * (input[i] > (792+offset0));

int rt0_0_199 = (input[i] > (797+offset0));
int t0_0_199 = rt0_0_199 * (2 + (input[i] > (798+offset0)));
t0_0_199 += (1 - rt0_0_199) * (input[i] > (796+offset0));

int rt0_0_200 = (input[i] > (801+offset0));
int t0_0_200 = rt0_0_200 * (2 + (input[i] > (802+offset0)));
t0_0_200 += (1 - rt0_0_200) * (input[i] > (800+offset0));

int rt0_0_201 = (input[i] > (805+offset0));
int t0_0_201 = rt0_0_201 * (2 + (input[i] > (806+offset0)));
t0_0_201 += (1 - rt0_0_201) * (input[i] > (804+offset0));

int rt0_0_202 = (input[i] > (809+offset0));
int t0_0_202 = rt0_0_202 * (2 + (input[i] > (810+offset0)));
t0_0_202 += (1 - rt0_0_202) * (input[i] > (808+offset0));

int rt0_0_203 = (input[i] > (813+offset0));
int t0_0_203 = rt0_0_203 * (2 + (input[i] > (814+offset0)));
t0_0_203 += (1 - rt0_0_203) * (input[i] > (812+offset0));

int rt0_0_204 = (input[i] > (817+offset0));
int t0_0_204 = rt0_0_204 * (2 + (input[i] > (818+offset0)));
t0_0_204 += (1 - rt0_0_204) * (input[i] > (816+offset0));

int rt0_0_205 = (input[i] > (821+offset0));
int t0_0_205 = rt0_0_205 * (2 + (input[i] > (822+offset0)));
t0_0_205 += (1 - rt0_0_205) * (input[i] > (820+offset0));

int rt0_0_206 = (input[i] > (825+offset0));
int t0_0_206 = rt0_0_206 * (2 + (input[i] > (826+offset0)));
t0_0_206 += (1 - rt0_0_206) * (input[i] > (824+offset0));

int rt0_0_207 = (input[i] > (829+offset0));
int t0_0_207 = rt0_0_207 * (2 + (input[i] > (830+offset0)));
t0_0_207 += (1 - rt0_0_207) * (input[i] > (828+offset0));

int rt0_0_208 = (input[i] > (833+offset0));
int t0_0_208 = rt0_0_208 * (2 + (input[i] > (834+offset0)));
t0_0_208 += (1 - rt0_0_208) * (input[i] > (832+offset0));

int rt0_0_209 = (input[i] > (837+offset0));
int t0_0_209 = rt0_0_209 * (2 + (input[i] > (838+offset0)));
t0_0_209 += (1 - rt0_0_209) * (input[i] > (836+offset0));

int rt0_0_210 = (input[i] > (841+offset0));
int t0_0_210 = rt0_0_210 * (2 + (input[i] > (842+offset0)));
t0_0_210 += (1 - rt0_0_210) * (input[i] > (840+offset0));

int rt0_0_211 = (input[i] > (845+offset0));
int t0_0_211 = rt0_0_211 * (2 + (input[i] > (846+offset0)));
t0_0_211 += (1 - rt0_0_211) * (input[i] > (844+offset0));

int rt0_0_212 = (input[i] > (849+offset0));
int t0_0_212 = rt0_0_212 * (2 + (input[i] > (850+offset0)));
t0_0_212 += (1 - rt0_0_212) * (input[i] > (848+offset0));

int rt0_0_213 = (input[i] > (853+offset0));
int t0_0_213 = rt0_0_213 * (2 + (input[i] > (854+offset0)));
t0_0_213 += (1 - rt0_0_213) * (input[i] > (852+offset0));

int rt0_0_214 = (input[i] > (857+offset0));
int t0_0_214 = rt0_0_214 * (2 + (input[i] > (858+offset0)));
t0_0_214 += (1 - rt0_0_214) * (input[i] > (856+offset0));

int rt0_0_215 = (input[i] > (861+offset0));
int t0_0_215 = rt0_0_215 * (2 + (input[i] > (862+offset0)));
t0_0_215 += (1 - rt0_0_215) * (input[i] > (860+offset0));

int rt0_0_216 = (input[i] > (865+offset0));
int t0_0_216 = rt0_0_216 * (2 + (input[i] > (866+offset0)));
t0_0_216 += (1 - rt0_0_216) * (input[i] > (864+offset0));

int rt0_0_217 = (input[i] > (869+offset0));
int t0_0_217 = rt0_0_217 * (2 + (input[i] > (870+offset0)));
t0_0_217 += (1 - rt0_0_217) * (input[i] > (868+offset0));

int rt0_0_218 = (input[i] > (873+offset0));
int t0_0_218 = rt0_0_218 * (2 + (input[i] > (874+offset0)));
t0_0_218 += (1 - rt0_0_218) * (input[i] > (872+offset0));

int rt0_0_219 = (input[i] > (877+offset0));
int t0_0_219 = rt0_0_219 * (2 + (input[i] > (878+offset0)));
t0_0_219 += (1 - rt0_0_219) * (input[i] > (876+offset0));

int rt0_0_220 = (input[i] > (881+offset0));
int t0_0_220 = rt0_0_220 * (2 + (input[i] > (882+offset0)));
t0_0_220 += (1 - rt0_0_220) * (input[i] > (880+offset0));

int rt0_0_221 = (input[i] > (885+offset0));
int t0_0_221 = rt0_0_221 * (2 + (input[i] > (886+offset0)));
t0_0_221 += (1 - rt0_0_221) * (input[i] > (884+offset0));

int rt0_0_222 = (input[i] > (889+offset0));
int t0_0_222 = rt0_0_222 * (2 + (input[i] > (890+offset0)));
t0_0_222 += (1 - rt0_0_222) * (input[i] > (888+offset0));

int rt0_0_223 = (input[i] > (893+offset0));
int t0_0_223 = rt0_0_223 * (2 + (input[i] > (894+offset0)));
t0_0_223 += (1 - rt0_0_223) * (input[i] > (892+offset0));

int rt0_0_224 = (input[i] > (897+offset0));
int t0_0_224 = rt0_0_224 * (2 + (input[i] > (898+offset0)));
t0_0_224 += (1 - rt0_0_224) * (input[i] > (896+offset0));

int rt0_0_225 = (input[i] > (901+offset0));
int t0_0_225 = rt0_0_225 * (2 + (input[i] > (902+offset0)));
t0_0_225 += (1 - rt0_0_225) * (input[i] > (900+offset0));

int rt0_0_226 = (input[i] > (905+offset0));
int t0_0_226 = rt0_0_226 * (2 + (input[i] > (906+offset0)));
t0_0_226 += (1 - rt0_0_226) * (input[i] > (904+offset0));

int rt0_0_227 = (input[i] > (909+offset0));
int t0_0_227 = rt0_0_227 * (2 + (input[i] > (910+offset0)));
t0_0_227 += (1 - rt0_0_227) * (input[i] > (908+offset0));

int rt0_0_228 = (input[i] > (913+offset0));
int t0_0_228 = rt0_0_228 * (2 + (input[i] > (914+offset0)));
t0_0_228 += (1 - rt0_0_228) * (input[i] > (912+offset0));

int rt0_0_229 = (input[i] > (917+offset0));
int t0_0_229 = rt0_0_229 * (2 + (input[i] > (918+offset0)));
t0_0_229 += (1 - rt0_0_229) * (input[i] > (916+offset0));

int rt0_0_230 = (input[i] > (921+offset0));
int t0_0_230 = rt0_0_230 * (2 + (input[i] > (922+offset0)));
t0_0_230 += (1 - rt0_0_230) * (input[i] > (920+offset0));

int rt0_0_231 = (input[i] > (925+offset0));
int t0_0_231 = rt0_0_231 * (2 + (input[i] > (926+offset0)));
t0_0_231 += (1 - rt0_0_231) * (input[i] > (924+offset0));

int rt0_0_232 = (input[i] > (929+offset0));
int t0_0_232 = rt0_0_232 * (2 + (input[i] > (930+offset0)));
t0_0_232 += (1 - rt0_0_232) * (input[i] > (928+offset0));

int rt0_0_233 = (input[i] > (933+offset0));
int t0_0_233 = rt0_0_233 * (2 + (input[i] > (934+offset0)));
t0_0_233 += (1 - rt0_0_233) * (input[i] > (932+offset0));

int rt0_0_234 = (input[i] > (937+offset0));
int t0_0_234 = rt0_0_234 * (2 + (input[i] > (938+offset0)));
t0_0_234 += (1 - rt0_0_234) * (input[i] > (936+offset0));

int rt0_0_235 = (input[i] > (941+offset0));
int t0_0_235 = rt0_0_235 * (2 + (input[i] > (942+offset0)));
t0_0_235 += (1 - rt0_0_235) * (input[i] > (940+offset0));

int rt0_0_236 = (input[i] > (945+offset0));
int t0_0_236 = rt0_0_236 * (2 + (input[i] > (946+offset0)));
t0_0_236 += (1 - rt0_0_236) * (input[i] > (944+offset0));

int rt0_0_237 = (input[i] > (949+offset0));
int t0_0_237 = rt0_0_237 * (2 + (input[i] > (950+offset0)));
t0_0_237 += (1 - rt0_0_237) * (input[i] > (948+offset0));

int rt0_0_238 = (input[i] > (953+offset0));
int t0_0_238 = rt0_0_238 * (2 + (input[i] > (954+offset0)));
t0_0_238 += (1 - rt0_0_238) * (input[i] > (952+offset0));

int rt0_0_239 = (input[i] > (957+offset0));
int t0_0_239 = rt0_0_239 * (2 + (input[i] > (958+offset0)));
t0_0_239 += (1 - rt0_0_239) * (input[i] > (956+offset0));

int rt0_0_240 = (input[i] > (961+offset0));
int t0_0_240 = rt0_0_240 * (2 + (input[i] > (962+offset0)));
t0_0_240 += (1 - rt0_0_240) * (input[i] > (960+offset0));

int rt0_0_241 = (input[i] > (965+offset0));
int t0_0_241 = rt0_0_241 * (2 + (input[i] > (966+offset0)));
t0_0_241 += (1 - rt0_0_241) * (input[i] > (964+offset0));

int rt0_0_242 = (input[i] > (969+offset0));
int t0_0_242 = rt0_0_242 * (2 + (input[i] > (970+offset0)));
t0_0_242 += (1 - rt0_0_242) * (input[i] > (968+offset0));

int rt0_0_243 = (input[i] > (973+offset0));
int t0_0_243 = rt0_0_243 * (2 + (input[i] > (974+offset0)));
t0_0_243 += (1 - rt0_0_243) * (input[i] > (972+offset0));

int rt0_0_244 = (input[i] > (977+offset0));
int t0_0_244 = rt0_0_244 * (2 + (input[i] > (978+offset0)));
t0_0_244 += (1 - rt0_0_244) * (input[i] > (976+offset0));

int rt0_0_245 = (input[i] > (981+offset0));
int t0_0_245 = rt0_0_245 * (2 + (input[i] > (982+offset0)));
t0_0_245 += (1 - rt0_0_245) * (input[i] > (980+offset0));

int rt0_0_246 = (input[i] > (985+offset0));
int t0_0_246 = rt0_0_246 * (2 + (input[i] > (986+offset0)));
t0_0_246 += (1 - rt0_0_246) * (input[i] > (984+offset0));

int rt0_0_247 = (input[i] > (989+offset0));
int t0_0_247 = rt0_0_247 * (2 + (input[i] > (990+offset0)));
t0_0_247 += (1 - rt0_0_247) * (input[i] > (988+offset0));

int rt0_0_248 = (input[i] > (993+offset0));
int t0_0_248 = rt0_0_248 * (2 + (input[i] > (994+offset0)));
t0_0_248 += (1 - rt0_0_248) * (input[i] > (992+offset0));

int rt0_0_249 = (input[i] > (997+offset0));
int t0_0_249 = rt0_0_249 * (2 + (input[i] > (998+offset0)));
t0_0_249 += (1 - rt0_0_249) * (input[i] > (996+offset0));

int rt0_0_250 = (input[i] > (1001+offset0));
int t0_0_250 = rt0_0_250 * (2 + (input[i] > (1002+offset0)));
t0_0_250 += (1 - rt0_0_250) * (input[i] > (1000+offset0));

int rt0_0_251 = (input[i] > (1005+offset0));
int t0_0_251 = rt0_0_251 * (2 + (input[i] > (1006+offset0)));
t0_0_251 += (1 - rt0_0_251) * (input[i] > (1004+offset0));

int rt0_0_252 = (input[i] > (1009+offset0));
int t0_0_252 = rt0_0_252 * (2 + (input[i] > (1010+offset0)));
t0_0_252 += (1 - rt0_0_252) * (input[i] > (1008+offset0));

int rt0_0_253 = (input[i] > (1013+offset0));
int t0_0_253 = rt0_0_253 * (2 + (input[i] > (1014+offset0)));
t0_0_253 += (1 - rt0_0_253) * (input[i] > (1012+offset0));

int rt0_0_254 = (input[i] > (1017+offset0));
int t0_0_254 = rt0_0_254 * (2 + (input[i] > (1018+offset0)));
t0_0_254 += (1 - rt0_0_254) * (input[i] > (1016+offset0));

int rt0_0_255 = (input[i] > (1021+offset0));
int t0_0_255 = rt0_0_255 * (2 + (input[i] > (1022+offset0)));
t0_0_255 += (1 - rt0_0_255) * (input[i] > (1020+offset0));

int rt0_1_0 = (input[i] > (3+offset0));
int t0_1_0 = rt0_1_0 * (4 + t0_0_1);
t0_1_0 += (1 - rt0_1_0) * t0_0_0;

int rt0_1_1 = (input[i] > (11+offset0));
int t0_1_1 = rt0_1_1 * (4 + t0_0_3);
t0_1_1 += (1 - rt0_1_1) * t0_0_2;

int rt0_1_2 = (input[i] > (19+offset0));
int t0_1_2 = rt0_1_2 * (4 + t0_0_5);
t0_1_2 += (1 - rt0_1_2) * t0_0_4;

int rt0_1_3 = (input[i] > (27+offset0));
int t0_1_3 = rt0_1_3 * (4 + t0_0_7);
t0_1_3 += (1 - rt0_1_3) * t0_0_6;

int rt0_1_4 = (input[i] > (35+offset0));
int t0_1_4 = rt0_1_4 * (4 + t0_0_9);
t0_1_4 += (1 - rt0_1_4) * t0_0_8;

int rt0_1_5 = (input[i] > (43+offset0));
int t0_1_5 = rt0_1_5 * (4 + t0_0_11);
t0_1_5 += (1 - rt0_1_5) * t0_0_10;

int rt0_1_6 = (input[i] > (51+offset0));
int t0_1_6 = rt0_1_6 * (4 + t0_0_13);
t0_1_6 += (1 - rt0_1_6) * t0_0_12;

int rt0_1_7 = (input[i] > (59+offset0));
int t0_1_7 = rt0_1_7 * (4 + t0_0_15);
t0_1_7 += (1 - rt0_1_7) * t0_0_14;

int rt0_1_8 = (input[i] > (67+offset0));
int t0_1_8 = rt0_1_8 * (4 + t0_0_17);
t0_1_8 += (1 - rt0_1_8) * t0_0_16;

int rt0_1_9 = (input[i] > (75+offset0));
int t0_1_9 = rt0_1_9 * (4 + t0_0_19);
t0_1_9 += (1 - rt0_1_9) * t0_0_18;

int rt0_1_10 = (input[i] > (83+offset0));
int t0_1_10 = rt0_1_10 * (4 + t0_0_21);
t0_1_10 += (1 - rt0_1_10) * t0_0_20;

int rt0_1_11 = (input[i] > (91+offset0));
int t0_1_11 = rt0_1_11 * (4 + t0_0_23);
t0_1_11 += (1 - rt0_1_11) * t0_0_22;

int rt0_1_12 = (input[i] > (99+offset0));
int t0_1_12 = rt0_1_12 * (4 + t0_0_25);
t0_1_12 += (1 - rt0_1_12) * t0_0_24;

int rt0_1_13 = (input[i] > (107+offset0));
int t0_1_13 = rt0_1_13 * (4 + t0_0_27);
t0_1_13 += (1 - rt0_1_13) * t0_0_26;

int rt0_1_14 = (input[i] > (115+offset0));
int t0_1_14 = rt0_1_14 * (4 + t0_0_29);
t0_1_14 += (1 - rt0_1_14) * t0_0_28;

int rt0_1_15 = (input[i] > (123+offset0));
int t0_1_15 = rt0_1_15 * (4 + t0_0_31);
t0_1_15 += (1 - rt0_1_15) * t0_0_30;

int rt0_1_16 = (input[i] > (131+offset0));
int t0_1_16 = rt0_1_16 * (4 + t0_0_33);
t0_1_16 += (1 - rt0_1_16) * t0_0_32;

int rt0_1_17 = (input[i] > (139+offset0));
int t0_1_17 = rt0_1_17 * (4 + t0_0_35);
t0_1_17 += (1 - rt0_1_17) * t0_0_34;

int rt0_1_18 = (input[i] > (147+offset0));
int t0_1_18 = rt0_1_18 * (4 + t0_0_37);
t0_1_18 += (1 - rt0_1_18) * t0_0_36;

int rt0_1_19 = (input[i] > (155+offset0));
int t0_1_19 = rt0_1_19 * (4 + t0_0_39);
t0_1_19 += (1 - rt0_1_19) * t0_0_38;

int rt0_1_20 = (input[i] > (163+offset0));
int t0_1_20 = rt0_1_20 * (4 + t0_0_41);
t0_1_20 += (1 - rt0_1_20) * t0_0_40;

int rt0_1_21 = (input[i] > (171+offset0));
int t0_1_21 = rt0_1_21 * (4 + t0_0_43);
t0_1_21 += (1 - rt0_1_21) * t0_0_42;

int rt0_1_22 = (input[i] > (179+offset0));
int t0_1_22 = rt0_1_22 * (4 + t0_0_45);
t0_1_22 += (1 - rt0_1_22) * t0_0_44;

int rt0_1_23 = (input[i] > (187+offset0));
int t0_1_23 = rt0_1_23 * (4 + t0_0_47);
t0_1_23 += (1 - rt0_1_23) * t0_0_46;

int rt0_1_24 = (input[i] > (195+offset0));
int t0_1_24 = rt0_1_24 * (4 + t0_0_49);
t0_1_24 += (1 - rt0_1_24) * t0_0_48;

int rt0_1_25 = (input[i] > (203+offset0));
int t0_1_25 = rt0_1_25 * (4 + t0_0_51);
t0_1_25 += (1 - rt0_1_25) * t0_0_50;

int rt0_1_26 = (input[i] > (211+offset0));
int t0_1_26 = rt0_1_26 * (4 + t0_0_53);
t0_1_26 += (1 - rt0_1_26) * t0_0_52;

int rt0_1_27 = (input[i] > (219+offset0));
int t0_1_27 = rt0_1_27 * (4 + t0_0_55);
t0_1_27 += (1 - rt0_1_27) * t0_0_54;

int rt0_1_28 = (input[i] > (227+offset0));
int t0_1_28 = rt0_1_28 * (4 + t0_0_57);
t0_1_28 += (1 - rt0_1_28) * t0_0_56;

int rt0_1_29 = (input[i] > (235+offset0));
int t0_1_29 = rt0_1_29 * (4 + t0_0_59);
t0_1_29 += (1 - rt0_1_29) * t0_0_58;

int rt0_1_30 = (input[i] > (243+offset0));
int t0_1_30 = rt0_1_30 * (4 + t0_0_61);
t0_1_30 += (1 - rt0_1_30) * t0_0_60;

int rt0_1_31 = (input[i] > (251+offset0));
int t0_1_31 = rt0_1_31 * (4 + t0_0_63);
t0_1_31 += (1 - rt0_1_31) * t0_0_62;

int rt0_1_32 = (input[i] > (259+offset0));
int t0_1_32 = rt0_1_32 * (4 + t0_0_65);
t0_1_32 += (1 - rt0_1_32) * t0_0_64;

int rt0_1_33 = (input[i] > (267+offset0));
int t0_1_33 = rt0_1_33 * (4 + t0_0_67);
t0_1_33 += (1 - rt0_1_33) * t0_0_66;

int rt0_1_34 = (input[i] > (275+offset0));
int t0_1_34 = rt0_1_34 * (4 + t0_0_69);
t0_1_34 += (1 - rt0_1_34) * t0_0_68;

int rt0_1_35 = (input[i] > (283+offset0));
int t0_1_35 = rt0_1_35 * (4 + t0_0_71);
t0_1_35 += (1 - rt0_1_35) * t0_0_70;

int rt0_1_36 = (input[i] > (291+offset0));
int t0_1_36 = rt0_1_36 * (4 + t0_0_73);
t0_1_36 += (1 - rt0_1_36) * t0_0_72;

int rt0_1_37 = (input[i] > (299+offset0));
int t0_1_37 = rt0_1_37 * (4 + t0_0_75);
t0_1_37 += (1 - rt0_1_37) * t0_0_74;

int rt0_1_38 = (input[i] > (307+offset0));
int t0_1_38 = rt0_1_38 * (4 + t0_0_77);
t0_1_38 += (1 - rt0_1_38) * t0_0_76;

int rt0_1_39 = (input[i] > (315+offset0));
int t0_1_39 = rt0_1_39 * (4 + t0_0_79);
t0_1_39 += (1 - rt0_1_39) * t0_0_78;

int rt0_1_40 = (input[i] > (323+offset0));
int t0_1_40 = rt0_1_40 * (4 + t0_0_81);
t0_1_40 += (1 - rt0_1_40) * t0_0_80;

int rt0_1_41 = (input[i] > (331+offset0));
int t0_1_41 = rt0_1_41 * (4 + t0_0_83);
t0_1_41 += (1 - rt0_1_41) * t0_0_82;

int rt0_1_42 = (input[i] > (339+offset0));
int t0_1_42 = rt0_1_42 * (4 + t0_0_85);
t0_1_42 += (1 - rt0_1_42) * t0_0_84;

int rt0_1_43 = (input[i] > (347+offset0));
int t0_1_43 = rt0_1_43 * (4 + t0_0_87);
t0_1_43 += (1 - rt0_1_43) * t0_0_86;

int rt0_1_44 = (input[i] > (355+offset0));
int t0_1_44 = rt0_1_44 * (4 + t0_0_89);
t0_1_44 += (1 - rt0_1_44) * t0_0_88;

int rt0_1_45 = (input[i] > (363+offset0));
int t0_1_45 = rt0_1_45 * (4 + t0_0_91);
t0_1_45 += (1 - rt0_1_45) * t0_0_90;

int rt0_1_46 = (input[i] > (371+offset0));
int t0_1_46 = rt0_1_46 * (4 + t0_0_93);
t0_1_46 += (1 - rt0_1_46) * t0_0_92;

int rt0_1_47 = (input[i] > (379+offset0));
int t0_1_47 = rt0_1_47 * (4 + t0_0_95);
t0_1_47 += (1 - rt0_1_47) * t0_0_94;

int rt0_1_48 = (input[i] > (387+offset0));
int t0_1_48 = rt0_1_48 * (4 + t0_0_97);
t0_1_48 += (1 - rt0_1_48) * t0_0_96;

int rt0_1_49 = (input[i] > (395+offset0));
int t0_1_49 = rt0_1_49 * (4 + t0_0_99);
t0_1_49 += (1 - rt0_1_49) * t0_0_98;

int rt0_1_50 = (input[i] > (403+offset0));
int t0_1_50 = rt0_1_50 * (4 + t0_0_101);
t0_1_50 += (1 - rt0_1_50) * t0_0_100;

int rt0_1_51 = (input[i] > (411+offset0));
int t0_1_51 = rt0_1_51 * (4 + t0_0_103);
t0_1_51 += (1 - rt0_1_51) * t0_0_102;

int rt0_1_52 = (input[i] > (419+offset0));
int t0_1_52 = rt0_1_52 * (4 + t0_0_105);
t0_1_52 += (1 - rt0_1_52) * t0_0_104;

int rt0_1_53 = (input[i] > (427+offset0));
int t0_1_53 = rt0_1_53 * (4 + t0_0_107);
t0_1_53 += (1 - rt0_1_53) * t0_0_106;

int rt0_1_54 = (input[i] > (435+offset0));
int t0_1_54 = rt0_1_54 * (4 + t0_0_109);
t0_1_54 += (1 - rt0_1_54) * t0_0_108;

int rt0_1_55 = (input[i] > (443+offset0));
int t0_1_55 = rt0_1_55 * (4 + t0_0_111);
t0_1_55 += (1 - rt0_1_55) * t0_0_110;

int rt0_1_56 = (input[i] > (451+offset0));
int t0_1_56 = rt0_1_56 * (4 + t0_0_113);
t0_1_56 += (1 - rt0_1_56) * t0_0_112;

int rt0_1_57 = (input[i] > (459+offset0));
int t0_1_57 = rt0_1_57 * (4 + t0_0_115);
t0_1_57 += (1 - rt0_1_57) * t0_0_114;

int rt0_1_58 = (input[i] > (467+offset0));
int t0_1_58 = rt0_1_58 * (4 + t0_0_117);
t0_1_58 += (1 - rt0_1_58) * t0_0_116;

int rt0_1_59 = (input[i] > (475+offset0));
int t0_1_59 = rt0_1_59 * (4 + t0_0_119);
t0_1_59 += (1 - rt0_1_59) * t0_0_118;

int rt0_1_60 = (input[i] > (483+offset0));
int t0_1_60 = rt0_1_60 * (4 + t0_0_121);
t0_1_60 += (1 - rt0_1_60) * t0_0_120;

int rt0_1_61 = (input[i] > (491+offset0));
int t0_1_61 = rt0_1_61 * (4 + t0_0_123);
t0_1_61 += (1 - rt0_1_61) * t0_0_122;

int rt0_1_62 = (input[i] > (499+offset0));
int t0_1_62 = rt0_1_62 * (4 + t0_0_125);
t0_1_62 += (1 - rt0_1_62) * t0_0_124;

int rt0_1_63 = (input[i] > (507+offset0));
int t0_1_63 = rt0_1_63 * (4 + t0_0_127);
t0_1_63 += (1 - rt0_1_63) * t0_0_126;

int rt0_1_64 = (input[i] > (515+offset0));
int t0_1_64 = rt0_1_64 * (4 + t0_0_129);
t0_1_64 += (1 - rt0_1_64) * t0_0_128;

int rt0_1_65 = (input[i] > (523+offset0));
int t0_1_65 = rt0_1_65 * (4 + t0_0_131);
t0_1_65 += (1 - rt0_1_65) * t0_0_130;

int rt0_1_66 = (input[i] > (531+offset0));
int t0_1_66 = rt0_1_66 * (4 + t0_0_133);
t0_1_66 += (1 - rt0_1_66) * t0_0_132;

int rt0_1_67 = (input[i] > (539+offset0));
int t0_1_67 = rt0_1_67 * (4 + t0_0_135);
t0_1_67 += (1 - rt0_1_67) * t0_0_134;

int rt0_1_68 = (input[i] > (547+offset0));
int t0_1_68 = rt0_1_68 * (4 + t0_0_137);
t0_1_68 += (1 - rt0_1_68) * t0_0_136;

int rt0_1_69 = (input[i] > (555+offset0));
int t0_1_69 = rt0_1_69 * (4 + t0_0_139);
t0_1_69 += (1 - rt0_1_69) * t0_0_138;

int rt0_1_70 = (input[i] > (563+offset0));
int t0_1_70 = rt0_1_70 * (4 + t0_0_141);
t0_1_70 += (1 - rt0_1_70) * t0_0_140;

int rt0_1_71 = (input[i] > (571+offset0));
int t0_1_71 = rt0_1_71 * (4 + t0_0_143);
t0_1_71 += (1 - rt0_1_71) * t0_0_142;

int rt0_1_72 = (input[i] > (579+offset0));
int t0_1_72 = rt0_1_72 * (4 + t0_0_145);
t0_1_72 += (1 - rt0_1_72) * t0_0_144;

int rt0_1_73 = (input[i] > (587+offset0));
int t0_1_73 = rt0_1_73 * (4 + t0_0_147);
t0_1_73 += (1 - rt0_1_73) * t0_0_146;

int rt0_1_74 = (input[i] > (595+offset0));
int t0_1_74 = rt0_1_74 * (4 + t0_0_149);
t0_1_74 += (1 - rt0_1_74) * t0_0_148;

int rt0_1_75 = (input[i] > (603+offset0));
int t0_1_75 = rt0_1_75 * (4 + t0_0_151);
t0_1_75 += (1 - rt0_1_75) * t0_0_150;

int rt0_1_76 = (input[i] > (611+offset0));
int t0_1_76 = rt0_1_76 * (4 + t0_0_153);
t0_1_76 += (1 - rt0_1_76) * t0_0_152;

int rt0_1_77 = (input[i] > (619+offset0));
int t0_1_77 = rt0_1_77 * (4 + t0_0_155);
t0_1_77 += (1 - rt0_1_77) * t0_0_154;

int rt0_1_78 = (input[i] > (627+offset0));
int t0_1_78 = rt0_1_78 * (4 + t0_0_157);
t0_1_78 += (1 - rt0_1_78) * t0_0_156;

int rt0_1_79 = (input[i] > (635+offset0));
int t0_1_79 = rt0_1_79 * (4 + t0_0_159);
t0_1_79 += (1 - rt0_1_79) * t0_0_158;

int rt0_1_80 = (input[i] > (643+offset0));
int t0_1_80 = rt0_1_80 * (4 + t0_0_161);
t0_1_80 += (1 - rt0_1_80) * t0_0_160;

int rt0_1_81 = (input[i] > (651+offset0));
int t0_1_81 = rt0_1_81 * (4 + t0_0_163);
t0_1_81 += (1 - rt0_1_81) * t0_0_162;

int rt0_1_82 = (input[i] > (659+offset0));
int t0_1_82 = rt0_1_82 * (4 + t0_0_165);
t0_1_82 += (1 - rt0_1_82) * t0_0_164;

int rt0_1_83 = (input[i] > (667+offset0));
int t0_1_83 = rt0_1_83 * (4 + t0_0_167);
t0_1_83 += (1 - rt0_1_83) * t0_0_166;

int rt0_1_84 = (input[i] > (675+offset0));
int t0_1_84 = rt0_1_84 * (4 + t0_0_169);
t0_1_84 += (1 - rt0_1_84) * t0_0_168;

int rt0_1_85 = (input[i] > (683+offset0));
int t0_1_85 = rt0_1_85 * (4 + t0_0_171);
t0_1_85 += (1 - rt0_1_85) * t0_0_170;

int rt0_1_86 = (input[i] > (691+offset0));
int t0_1_86 = rt0_1_86 * (4 + t0_0_173);
t0_1_86 += (1 - rt0_1_86) * t0_0_172;

int rt0_1_87 = (input[i] > (699+offset0));
int t0_1_87 = rt0_1_87 * (4 + t0_0_175);
t0_1_87 += (1 - rt0_1_87) * t0_0_174;

int rt0_1_88 = (input[i] > (707+offset0));
int t0_1_88 = rt0_1_88 * (4 + t0_0_177);
t0_1_88 += (1 - rt0_1_88) * t0_0_176;

int rt0_1_89 = (input[i] > (715+offset0));
int t0_1_89 = rt0_1_89 * (4 + t0_0_179);
t0_1_89 += (1 - rt0_1_89) * t0_0_178;

int rt0_1_90 = (input[i] > (723+offset0));
int t0_1_90 = rt0_1_90 * (4 + t0_0_181);
t0_1_90 += (1 - rt0_1_90) * t0_0_180;

int rt0_1_91 = (input[i] > (731+offset0));
int t0_1_91 = rt0_1_91 * (4 + t0_0_183);
t0_1_91 += (1 - rt0_1_91) * t0_0_182;

int rt0_1_92 = (input[i] > (739+offset0));
int t0_1_92 = rt0_1_92 * (4 + t0_0_185);
t0_1_92 += (1 - rt0_1_92) * t0_0_184;

int rt0_1_93 = (input[i] > (747+offset0));
int t0_1_93 = rt0_1_93 * (4 + t0_0_187);
t0_1_93 += (1 - rt0_1_93) * t0_0_186;

int rt0_1_94 = (input[i] > (755+offset0));
int t0_1_94 = rt0_1_94 * (4 + t0_0_189);
t0_1_94 += (1 - rt0_1_94) * t0_0_188;

int rt0_1_95 = (input[i] > (763+offset0));
int t0_1_95 = rt0_1_95 * (4 + t0_0_191);
t0_1_95 += (1 - rt0_1_95) * t0_0_190;

int rt0_1_96 = (input[i] > (771+offset0));
int t0_1_96 = rt0_1_96 * (4 + t0_0_193);
t0_1_96 += (1 - rt0_1_96) * t0_0_192;

int rt0_1_97 = (input[i] > (779+offset0));
int t0_1_97 = rt0_1_97 * (4 + t0_0_195);
t0_1_97 += (1 - rt0_1_97) * t0_0_194;

int rt0_1_98 = (input[i] > (787+offset0));
int t0_1_98 = rt0_1_98 * (4 + t0_0_197);
t0_1_98 += (1 - rt0_1_98) * t0_0_196;

int rt0_1_99 = (input[i] > (795+offset0));
int t0_1_99 = rt0_1_99 * (4 + t0_0_199);
t0_1_99 += (1 - rt0_1_99) * t0_0_198;

int rt0_1_100 = (input[i] > (803+offset0));
int t0_1_100 = rt0_1_100 * (4 + t0_0_201);
t0_1_100 += (1 - rt0_1_100) * t0_0_200;

int rt0_1_101 = (input[i] > (811+offset0));
int t0_1_101 = rt0_1_101 * (4 + t0_0_203);
t0_1_101 += (1 - rt0_1_101) * t0_0_202;

int rt0_1_102 = (input[i] > (819+offset0));
int t0_1_102 = rt0_1_102 * (4 + t0_0_205);
t0_1_102 += (1 - rt0_1_102) * t0_0_204;

int rt0_1_103 = (input[i] > (827+offset0));
int t0_1_103 = rt0_1_103 * (4 + t0_0_207);
t0_1_103 += (1 - rt0_1_103) * t0_0_206;

int rt0_1_104 = (input[i] > (835+offset0));
int t0_1_104 = rt0_1_104 * (4 + t0_0_209);
t0_1_104 += (1 - rt0_1_104) * t0_0_208;

int rt0_1_105 = (input[i] > (843+offset0));
int t0_1_105 = rt0_1_105 * (4 + t0_0_211);
t0_1_105 += (1 - rt0_1_105) * t0_0_210;

int rt0_1_106 = (input[i] > (851+offset0));
int t0_1_106 = rt0_1_106 * (4 + t0_0_213);
t0_1_106 += (1 - rt0_1_106) * t0_0_212;

int rt0_1_107 = (input[i] > (859+offset0));
int t0_1_107 = rt0_1_107 * (4 + t0_0_215);
t0_1_107 += (1 - rt0_1_107) * t0_0_214;

int rt0_1_108 = (input[i] > (867+offset0));
int t0_1_108 = rt0_1_108 * (4 + t0_0_217);
t0_1_108 += (1 - rt0_1_108) * t0_0_216;

int rt0_1_109 = (input[i] > (875+offset0));
int t0_1_109 = rt0_1_109 * (4 + t0_0_219);
t0_1_109 += (1 - rt0_1_109) * t0_0_218;

int rt0_1_110 = (input[i] > (883+offset0));
int t0_1_110 = rt0_1_110 * (4 + t0_0_221);
t0_1_110 += (1 - rt0_1_110) * t0_0_220;

int rt0_1_111 = (input[i] > (891+offset0));
int t0_1_111 = rt0_1_111 * (4 + t0_0_223);
t0_1_111 += (1 - rt0_1_111) * t0_0_222;

int rt0_1_112 = (input[i] > (899+offset0));
int t0_1_112 = rt0_1_112 * (4 + t0_0_225);
t0_1_112 += (1 - rt0_1_112) * t0_0_224;

int rt0_1_113 = (input[i] > (907+offset0));
int t0_1_113 = rt0_1_113 * (4 + t0_0_227);
t0_1_113 += (1 - rt0_1_113) * t0_0_226;

int rt0_1_114 = (input[i] > (915+offset0));
int t0_1_114 = rt0_1_114 * (4 + t0_0_229);
t0_1_114 += (1 - rt0_1_114) * t0_0_228;

int rt0_1_115 = (input[i] > (923+offset0));
int t0_1_115 = rt0_1_115 * (4 + t0_0_231);
t0_1_115 += (1 - rt0_1_115) * t0_0_230;

int rt0_1_116 = (input[i] > (931+offset0));
int t0_1_116 = rt0_1_116 * (4 + t0_0_233);
t0_1_116 += (1 - rt0_1_116) * t0_0_232;

int rt0_1_117 = (input[i] > (939+offset0));
int t0_1_117 = rt0_1_117 * (4 + t0_0_235);
t0_1_117 += (1 - rt0_1_117) * t0_0_234;

int rt0_1_118 = (input[i] > (947+offset0));
int t0_1_118 = rt0_1_118 * (4 + t0_0_237);
t0_1_118 += (1 - rt0_1_118) * t0_0_236;

int rt0_1_119 = (input[i] > (955+offset0));
int t0_1_119 = rt0_1_119 * (4 + t0_0_239);
t0_1_119 += (1 - rt0_1_119) * t0_0_238;

int rt0_1_120 = (input[i] > (963+offset0));
int t0_1_120 = rt0_1_120 * (4 + t0_0_241);
t0_1_120 += (1 - rt0_1_120) * t0_0_240;

int rt0_1_121 = (input[i] > (971+offset0));
int t0_1_121 = rt0_1_121 * (4 + t0_0_243);
t0_1_121 += (1 - rt0_1_121) * t0_0_242;

int rt0_1_122 = (input[i] > (979+offset0));
int t0_1_122 = rt0_1_122 * (4 + t0_0_245);
t0_1_122 += (1 - rt0_1_122) * t0_0_244;

int rt0_1_123 = (input[i] > (987+offset0));
int t0_1_123 = rt0_1_123 * (4 + t0_0_247);
t0_1_123 += (1 - rt0_1_123) * t0_0_246;

int rt0_1_124 = (input[i] > (995+offset0));
int t0_1_124 = rt0_1_124 * (4 + t0_0_249);
t0_1_124 += (1 - rt0_1_124) * t0_0_248;

int rt0_1_125 = (input[i] > (1003+offset0));
int t0_1_125 = rt0_1_125 * (4 + t0_0_251);
t0_1_125 += (1 - rt0_1_125) * t0_0_250;

int rt0_1_126 = (input[i] > (1011+offset0));
int t0_1_126 = rt0_1_126 * (4 + t0_0_253);
t0_1_126 += (1 - rt0_1_126) * t0_0_252;

int rt0_1_127 = (input[i] > (1019+offset0));
int t0_1_127 = rt0_1_127 * (4 + t0_0_255);
t0_1_127 += (1 - rt0_1_127) * t0_0_254;

int rt0_2_0 = (input[i] > (7+offset0));
int t0_2_0 = rt0_2_0 * (8 + t0_1_1);
t0_2_0 += (1 - rt0_2_0) * t0_1_0;

int rt0_2_1 = (input[i] > (23+offset0));
int t0_2_1 = rt0_2_1 * (8 + t0_1_3);
t0_2_1 += (1 - rt0_2_1) * t0_1_2;

int rt0_2_2 = (input[i] > (39+offset0));
int t0_2_2 = rt0_2_2 * (8 + t0_1_5);
t0_2_2 += (1 - rt0_2_2) * t0_1_4;

int rt0_2_3 = (input[i] > (55+offset0));
int t0_2_3 = rt0_2_3 * (8 + t0_1_7);
t0_2_3 += (1 - rt0_2_3) * t0_1_6;

int rt0_2_4 = (input[i] > (71+offset0));
int t0_2_4 = rt0_2_4 * (8 + t0_1_9);
t0_2_4 += (1 - rt0_2_4) * t0_1_8;

int rt0_2_5 = (input[i] > (87+offset0));
int t0_2_5 = rt0_2_5 * (8 + t0_1_11);
t0_2_5 += (1 - rt0_2_5) * t0_1_10;

int rt0_2_6 = (input[i] > (103+offset0));
int t0_2_6 = rt0_2_6 * (8 + t0_1_13);
t0_2_6 += (1 - rt0_2_6) * t0_1_12;

int rt0_2_7 = (input[i] > (119+offset0));
int t0_2_7 = rt0_2_7 * (8 + t0_1_15);
t0_2_7 += (1 - rt0_2_7) * t0_1_14;

int rt0_2_8 = (input[i] > (135+offset0));
int t0_2_8 = rt0_2_8 * (8 + t0_1_17);
t0_2_8 += (1 - rt0_2_8) * t0_1_16;

int rt0_2_9 = (input[i] > (151+offset0));
int t0_2_9 = rt0_2_9 * (8 + t0_1_19);
t0_2_9 += (1 - rt0_2_9) * t0_1_18;

int rt0_2_10 = (input[i] > (167+offset0));
int t0_2_10 = rt0_2_10 * (8 + t0_1_21);
t0_2_10 += (1 - rt0_2_10) * t0_1_20;

int rt0_2_11 = (input[i] > (183+offset0));
int t0_2_11 = rt0_2_11 * (8 + t0_1_23);
t0_2_11 += (1 - rt0_2_11) * t0_1_22;

int rt0_2_12 = (input[i] > (199+offset0));
int t0_2_12 = rt0_2_12 * (8 + t0_1_25);
t0_2_12 += (1 - rt0_2_12) * t0_1_24;

int rt0_2_13 = (input[i] > (215+offset0));
int t0_2_13 = rt0_2_13 * (8 + t0_1_27);
t0_2_13 += (1 - rt0_2_13) * t0_1_26;

int rt0_2_14 = (input[i] > (231+offset0));
int t0_2_14 = rt0_2_14 * (8 + t0_1_29);
t0_2_14 += (1 - rt0_2_14) * t0_1_28;

int rt0_2_15 = (input[i] > (247+offset0));
int t0_2_15 = rt0_2_15 * (8 + t0_1_31);
t0_2_15 += (1 - rt0_2_15) * t0_1_30;

int rt0_2_16 = (input[i] > (263+offset0));
int t0_2_16 = rt0_2_16 * (8 + t0_1_33);
t0_2_16 += (1 - rt0_2_16) * t0_1_32;

int rt0_2_17 = (input[i] > (279+offset0));
int t0_2_17 = rt0_2_17 * (8 + t0_1_35);
t0_2_17 += (1 - rt0_2_17) * t0_1_34;

int rt0_2_18 = (input[i] > (295+offset0));
int t0_2_18 = rt0_2_18 * (8 + t0_1_37);
t0_2_18 += (1 - rt0_2_18) * t0_1_36;

int rt0_2_19 = (input[i] > (311+offset0));
int t0_2_19 = rt0_2_19 * (8 + t0_1_39);
t0_2_19 += (1 - rt0_2_19) * t0_1_38;

int rt0_2_20 = (input[i] > (327+offset0));
int t0_2_20 = rt0_2_20 * (8 + t0_1_41);
t0_2_20 += (1 - rt0_2_20) * t0_1_40;

int rt0_2_21 = (input[i] > (343+offset0));
int t0_2_21 = rt0_2_21 * (8 + t0_1_43);
t0_2_21 += (1 - rt0_2_21) * t0_1_42;

int rt0_2_22 = (input[i] > (359+offset0));
int t0_2_22 = rt0_2_22 * (8 + t0_1_45);
t0_2_22 += (1 - rt0_2_22) * t0_1_44;

int rt0_2_23 = (input[i] > (375+offset0));
int t0_2_23 = rt0_2_23 * (8 + t0_1_47);
t0_2_23 += (1 - rt0_2_23) * t0_1_46;

int rt0_2_24 = (input[i] > (391+offset0));
int t0_2_24 = rt0_2_24 * (8 + t0_1_49);
t0_2_24 += (1 - rt0_2_24) * t0_1_48;

int rt0_2_25 = (input[i] > (407+offset0));
int t0_2_25 = rt0_2_25 * (8 + t0_1_51);
t0_2_25 += (1 - rt0_2_25) * t0_1_50;

int rt0_2_26 = (input[i] > (423+offset0));
int t0_2_26 = rt0_2_26 * (8 + t0_1_53);
t0_2_26 += (1 - rt0_2_26) * t0_1_52;

int rt0_2_27 = (input[i] > (439+offset0));
int t0_2_27 = rt0_2_27 * (8 + t0_1_55);
t0_2_27 += (1 - rt0_2_27) * t0_1_54;

int rt0_2_28 = (input[i] > (455+offset0));
int t0_2_28 = rt0_2_28 * (8 + t0_1_57);
t0_2_28 += (1 - rt0_2_28) * t0_1_56;

int rt0_2_29 = (input[i] > (471+offset0));
int t0_2_29 = rt0_2_29 * (8 + t0_1_59);
t0_2_29 += (1 - rt0_2_29) * t0_1_58;

int rt0_2_30 = (input[i] > (487+offset0));
int t0_2_30 = rt0_2_30 * (8 + t0_1_61);
t0_2_30 += (1 - rt0_2_30) * t0_1_60;

int rt0_2_31 = (input[i] > (503+offset0));
int t0_2_31 = rt0_2_31 * (8 + t0_1_63);
t0_2_31 += (1 - rt0_2_31) * t0_1_62;

int rt0_2_32 = (input[i] > (519+offset0));
int t0_2_32 = rt0_2_32 * (8 + t0_1_65);
t0_2_32 += (1 - rt0_2_32) * t0_1_64;

int rt0_2_33 = (input[i] > (535+offset0));
int t0_2_33 = rt0_2_33 * (8 + t0_1_67);
t0_2_33 += (1 - rt0_2_33) * t0_1_66;

int rt0_2_34 = (input[i] > (551+offset0));
int t0_2_34 = rt0_2_34 * (8 + t0_1_69);
t0_2_34 += (1 - rt0_2_34) * t0_1_68;

int rt0_2_35 = (input[i] > (567+offset0));
int t0_2_35 = rt0_2_35 * (8 + t0_1_71);
t0_2_35 += (1 - rt0_2_35) * t0_1_70;

int rt0_2_36 = (input[i] > (583+offset0));
int t0_2_36 = rt0_2_36 * (8 + t0_1_73);
t0_2_36 += (1 - rt0_2_36) * t0_1_72;

int rt0_2_37 = (input[i] > (599+offset0));
int t0_2_37 = rt0_2_37 * (8 + t0_1_75);
t0_2_37 += (1 - rt0_2_37) * t0_1_74;

int rt0_2_38 = (input[i] > (615+offset0));
int t0_2_38 = rt0_2_38 * (8 + t0_1_77);
t0_2_38 += (1 - rt0_2_38) * t0_1_76;

int rt0_2_39 = (input[i] > (631+offset0));
int t0_2_39 = rt0_2_39 * (8 + t0_1_79);
t0_2_39 += (1 - rt0_2_39) * t0_1_78;

int rt0_2_40 = (input[i] > (647+offset0));
int t0_2_40 = rt0_2_40 * (8 + t0_1_81);
t0_2_40 += (1 - rt0_2_40) * t0_1_80;

int rt0_2_41 = (input[i] > (663+offset0));
int t0_2_41 = rt0_2_41 * (8 + t0_1_83);
t0_2_41 += (1 - rt0_2_41) * t0_1_82;

int rt0_2_42 = (input[i] > (679+offset0));
int t0_2_42 = rt0_2_42 * (8 + t0_1_85);
t0_2_42 += (1 - rt0_2_42) * t0_1_84;

int rt0_2_43 = (input[i] > (695+offset0));
int t0_2_43 = rt0_2_43 * (8 + t0_1_87);
t0_2_43 += (1 - rt0_2_43) * t0_1_86;

int rt0_2_44 = (input[i] > (711+offset0));
int t0_2_44 = rt0_2_44 * (8 + t0_1_89);
t0_2_44 += (1 - rt0_2_44) * t0_1_88;

int rt0_2_45 = (input[i] > (727+offset0));
int t0_2_45 = rt0_2_45 * (8 + t0_1_91);
t0_2_45 += (1 - rt0_2_45) * t0_1_90;

int rt0_2_46 = (input[i] > (743+offset0));
int t0_2_46 = rt0_2_46 * (8 + t0_1_93);
t0_2_46 += (1 - rt0_2_46) * t0_1_92;

int rt0_2_47 = (input[i] > (759+offset0));
int t0_2_47 = rt0_2_47 * (8 + t0_1_95);
t0_2_47 += (1 - rt0_2_47) * t0_1_94;

int rt0_2_48 = (input[i] > (775+offset0));
int t0_2_48 = rt0_2_48 * (8 + t0_1_97);
t0_2_48 += (1 - rt0_2_48) * t0_1_96;

int rt0_2_49 = (input[i] > (791+offset0));
int t0_2_49 = rt0_2_49 * (8 + t0_1_99);
t0_2_49 += (1 - rt0_2_49) * t0_1_98;

int rt0_2_50 = (input[i] > (807+offset0));
int t0_2_50 = rt0_2_50 * (8 + t0_1_101);
t0_2_50 += (1 - rt0_2_50) * t0_1_100;

int rt0_2_51 = (input[i] > (823+offset0));
int t0_2_51 = rt0_2_51 * (8 + t0_1_103);
t0_2_51 += (1 - rt0_2_51) * t0_1_102;

int rt0_2_52 = (input[i] > (839+offset0));
int t0_2_52 = rt0_2_52 * (8 + t0_1_105);
t0_2_52 += (1 - rt0_2_52) * t0_1_104;

int rt0_2_53 = (input[i] > (855+offset0));
int t0_2_53 = rt0_2_53 * (8 + t0_1_107);
t0_2_53 += (1 - rt0_2_53) * t0_1_106;

int rt0_2_54 = (input[i] > (871+offset0));
int t0_2_54 = rt0_2_54 * (8 + t0_1_109);
t0_2_54 += (1 - rt0_2_54) * t0_1_108;

int rt0_2_55 = (input[i] > (887+offset0));
int t0_2_55 = rt0_2_55 * (8 + t0_1_111);
t0_2_55 += (1 - rt0_2_55) * t0_1_110;

int rt0_2_56 = (input[i] > (903+offset0));
int t0_2_56 = rt0_2_56 * (8 + t0_1_113);
t0_2_56 += (1 - rt0_2_56) * t0_1_112;

int rt0_2_57 = (input[i] > (919+offset0));
int t0_2_57 = rt0_2_57 * (8 + t0_1_115);
t0_2_57 += (1 - rt0_2_57) * t0_1_114;

int rt0_2_58 = (input[i] > (935+offset0));
int t0_2_58 = rt0_2_58 * (8 + t0_1_117);
t0_2_58 += (1 - rt0_2_58) * t0_1_116;

int rt0_2_59 = (input[i] > (951+offset0));
int t0_2_59 = rt0_2_59 * (8 + t0_1_119);
t0_2_59 += (1 - rt0_2_59) * t0_1_118;

int rt0_2_60 = (input[i] > (967+offset0));
int t0_2_60 = rt0_2_60 * (8 + t0_1_121);
t0_2_60 += (1 - rt0_2_60) * t0_1_120;

int rt0_2_61 = (input[i] > (983+offset0));
int t0_2_61 = rt0_2_61 * (8 + t0_1_123);
t0_2_61 += (1 - rt0_2_61) * t0_1_122;

int rt0_2_62 = (input[i] > (999+offset0));
int t0_2_62 = rt0_2_62 * (8 + t0_1_125);
t0_2_62 += (1 - rt0_2_62) * t0_1_124;

int rt0_2_63 = (input[i] > (1015+offset0));
int t0_2_63 = rt0_2_63 * (8 + t0_1_127);
t0_2_63 += (1 - rt0_2_63) * t0_1_126;

int rt0_3_0 = (input[i] > (15+offset0));
int t0_3_0 = rt0_3_0 * (16 + t0_2_1);
t0_3_0 += (1 - rt0_3_0) * t0_2_0;

int rt0_3_1 = (input[i] > (47+offset0));
int t0_3_1 = rt0_3_1 * (16 + t0_2_3);
t0_3_1 += (1 - rt0_3_1) * t0_2_2;

int rt0_3_2 = (input[i] > (79+offset0));
int t0_3_2 = rt0_3_2 * (16 + t0_2_5);
t0_3_2 += (1 - rt0_3_2) * t0_2_4;

int rt0_3_3 = (input[i] > (111+offset0));
int t0_3_3 = rt0_3_3 * (16 + t0_2_7);
t0_3_3 += (1 - rt0_3_3) * t0_2_6;

int rt0_3_4 = (input[i] > (143+offset0));
int t0_3_4 = rt0_3_4 * (16 + t0_2_9);
t0_3_4 += (1 - rt0_3_4) * t0_2_8;

int rt0_3_5 = (input[i] > (175+offset0));
int t0_3_5 = rt0_3_5 * (16 + t0_2_11);
t0_3_5 += (1 - rt0_3_5) * t0_2_10;

int rt0_3_6 = (input[i] > (207+offset0));
int t0_3_6 = rt0_3_6 * (16 + t0_2_13);
t0_3_6 += (1 - rt0_3_6) * t0_2_12;

int rt0_3_7 = (input[i] > (239+offset0));
int t0_3_7 = rt0_3_7 * (16 + t0_2_15);
t0_3_7 += (1 - rt0_3_7) * t0_2_14;

int rt0_3_8 = (input[i] > (271+offset0));
int t0_3_8 = rt0_3_8 * (16 + t0_2_17);
t0_3_8 += (1 - rt0_3_8) * t0_2_16;

int rt0_3_9 = (input[i] > (303+offset0));
int t0_3_9 = rt0_3_9 * (16 + t0_2_19);
t0_3_9 += (1 - rt0_3_9) * t0_2_18;

int rt0_3_10 = (input[i] > (335+offset0));
int t0_3_10 = rt0_3_10 * (16 + t0_2_21);
t0_3_10 += (1 - rt0_3_10) * t0_2_20;

int rt0_3_11 = (input[i] > (367+offset0));
int t0_3_11 = rt0_3_11 * (16 + t0_2_23);
t0_3_11 += (1 - rt0_3_11) * t0_2_22;

int rt0_3_12 = (input[i] > (399+offset0));
int t0_3_12 = rt0_3_12 * (16 + t0_2_25);
t0_3_12 += (1 - rt0_3_12) * t0_2_24;

int rt0_3_13 = (input[i] > (431+offset0));
int t0_3_13 = rt0_3_13 * (16 + t0_2_27);
t0_3_13 += (1 - rt0_3_13) * t0_2_26;

int rt0_3_14 = (input[i] > (463+offset0));
int t0_3_14 = rt0_3_14 * (16 + t0_2_29);
t0_3_14 += (1 - rt0_3_14) * t0_2_28;

int rt0_3_15 = (input[i] > (495+offset0));
int t0_3_15 = rt0_3_15 * (16 + t0_2_31);
t0_3_15 += (1 - rt0_3_15) * t0_2_30;

int rt0_3_16 = (input[i] > (527+offset0));
int t0_3_16 = rt0_3_16 * (16 + t0_2_33);
t0_3_16 += (1 - rt0_3_16) * t0_2_32;

int rt0_3_17 = (input[i] > (559+offset0));
int t0_3_17 = rt0_3_17 * (16 + t0_2_35);
t0_3_17 += (1 - rt0_3_17) * t0_2_34;

int rt0_3_18 = (input[i] > (591+offset0));
int t0_3_18 = rt0_3_18 * (16 + t0_2_37);
t0_3_18 += (1 - rt0_3_18) * t0_2_36;

int rt0_3_19 = (input[i] > (623+offset0));
int t0_3_19 = rt0_3_19 * (16 + t0_2_39);
t0_3_19 += (1 - rt0_3_19) * t0_2_38;

int rt0_3_20 = (input[i] > (655+offset0));
int t0_3_20 = rt0_3_20 * (16 + t0_2_41);
t0_3_20 += (1 - rt0_3_20) * t0_2_40;

int rt0_3_21 = (input[i] > (687+offset0));
int t0_3_21 = rt0_3_21 * (16 + t0_2_43);
t0_3_21 += (1 - rt0_3_21) * t0_2_42;

int rt0_3_22 = (input[i] > (719+offset0));
int t0_3_22 = rt0_3_22 * (16 + t0_2_45);
t0_3_22 += (1 - rt0_3_22) * t0_2_44;

int rt0_3_23 = (input[i] > (751+offset0));
int t0_3_23 = rt0_3_23 * (16 + t0_2_47);
t0_3_23 += (1 - rt0_3_23) * t0_2_46;

int rt0_3_24 = (input[i] > (783+offset0));
int t0_3_24 = rt0_3_24 * (16 + t0_2_49);
t0_3_24 += (1 - rt0_3_24) * t0_2_48;

int rt0_3_25 = (input[i] > (815+offset0));
int t0_3_25 = rt0_3_25 * (16 + t0_2_51);
t0_3_25 += (1 - rt0_3_25) * t0_2_50;

int rt0_3_26 = (input[i] > (847+offset0));
int t0_3_26 = rt0_3_26 * (16 + t0_2_53);
t0_3_26 += (1 - rt0_3_26) * t0_2_52;

int rt0_3_27 = (input[i] > (879+offset0));
int t0_3_27 = rt0_3_27 * (16 + t0_2_55);
t0_3_27 += (1 - rt0_3_27) * t0_2_54;

int rt0_3_28 = (input[i] > (911+offset0));
int t0_3_28 = rt0_3_28 * (16 + t0_2_57);
t0_3_28 += (1 - rt0_3_28) * t0_2_56;

int rt0_3_29 = (input[i] > (943+offset0));
int t0_3_29 = rt0_3_29 * (16 + t0_2_59);
t0_3_29 += (1 - rt0_3_29) * t0_2_58;

int rt0_3_30 = (input[i] > (975+offset0));
int t0_3_30 = rt0_3_30 * (16 + t0_2_61);
t0_3_30 += (1 - rt0_3_30) * t0_2_60;

int rt0_3_31 = (input[i] > (1007+offset0));
int t0_3_31 = rt0_3_31 * (16 + t0_2_63);
t0_3_31 += (1 - rt0_3_31) * t0_2_62;

int rt0_4_0 = (input[i] > (31+offset0));
int t0_4_0 = rt0_4_0 * (32 + t0_3_1);
t0_4_0 += (1 - rt0_4_0) * t0_3_0;

int rt0_4_1 = (input[i] > (95+offset0));
int t0_4_1 = rt0_4_1 * (32 + t0_3_3);
t0_4_1 += (1 - rt0_4_1) * t0_3_2;

int rt0_4_2 = (input[i] > (159+offset0));
int t0_4_2 = rt0_4_2 * (32 + t0_3_5);
t0_4_2 += (1 - rt0_4_2) * t0_3_4;

int rt0_4_3 = (input[i] > (223+offset0));
int t0_4_3 = rt0_4_3 * (32 + t0_3_7);
t0_4_3 += (1 - rt0_4_3) * t0_3_6;

int rt0_4_4 = (input[i] > (287+offset0));
int t0_4_4 = rt0_4_4 * (32 + t0_3_9);
t0_4_4 += (1 - rt0_4_4) * t0_3_8;

int rt0_4_5 = (input[i] > (351+offset0));
int t0_4_5 = rt0_4_5 * (32 + t0_3_11);
t0_4_5 += (1 - rt0_4_5) * t0_3_10;

int rt0_4_6 = (input[i] > (415+offset0));
int t0_4_6 = rt0_4_6 * (32 + t0_3_13);
t0_4_6 += (1 - rt0_4_6) * t0_3_12;

int rt0_4_7 = (input[i] > (479+offset0));
int t0_4_7 = rt0_4_7 * (32 + t0_3_15);
t0_4_7 += (1 - rt0_4_7) * t0_3_14;

int rt0_4_8 = (input[i] > (543+offset0));
int t0_4_8 = rt0_4_8 * (32 + t0_3_17);
t0_4_8 += (1 - rt0_4_8) * t0_3_16;

int rt0_4_9 = (input[i] > (607+offset0));
int t0_4_9 = rt0_4_9 * (32 + t0_3_19);
t0_4_9 += (1 - rt0_4_9) * t0_3_18;

int rt0_4_10 = (input[i] > (671+offset0));
int t0_4_10 = rt0_4_10 * (32 + t0_3_21);
t0_4_10 += (1 - rt0_4_10) * t0_3_20;

int rt0_4_11 = (input[i] > (735+offset0));
int t0_4_11 = rt0_4_11 * (32 + t0_3_23);
t0_4_11 += (1 - rt0_4_11) * t0_3_22;

int rt0_4_12 = (input[i] > (799+offset0));
int t0_4_12 = rt0_4_12 * (32 + t0_3_25);
t0_4_12 += (1 - rt0_4_12) * t0_3_24;

int rt0_4_13 = (input[i] > (863+offset0));
int t0_4_13 = rt0_4_13 * (32 + t0_3_27);
t0_4_13 += (1 - rt0_4_13) * t0_3_26;

int rt0_4_14 = (input[i] > (927+offset0));
int t0_4_14 = rt0_4_14 * (32 + t0_3_29);
t0_4_14 += (1 - rt0_4_14) * t0_3_28;

int rt0_4_15 = (input[i] > (991+offset0));
int t0_4_15 = rt0_4_15 * (32 + t0_3_31);
t0_4_15 += (1 - rt0_4_15) * t0_3_30;

int rt0_5_0 = (input[i] > (63+offset0));
int t0_5_0 = rt0_5_0 * (64 + t0_4_1);
t0_5_0 += (1 - rt0_5_0) * t0_4_0;

int rt0_5_1 = (input[i] > (191+offset0));
int t0_5_1 = rt0_5_1 * (64 + t0_4_3);
t0_5_1 += (1 - rt0_5_1) * t0_4_2;

int rt0_5_2 = (input[i] > (319+offset0));
int t0_5_2 = rt0_5_2 * (64 + t0_4_5);
t0_5_2 += (1 - rt0_5_2) * t0_4_4;

int rt0_5_3 = (input[i] > (447+offset0));
int t0_5_3 = rt0_5_3 * (64 + t0_4_7);
t0_5_3 += (1 - rt0_5_3) * t0_4_6;

int rt0_5_4 = (input[i] > (575+offset0));
int t0_5_4 = rt0_5_4 * (64 + t0_4_9);
t0_5_4 += (1 - rt0_5_4) * t0_4_8;

int rt0_5_5 = (input[i] > (703+offset0));
int t0_5_5 = rt0_5_5 * (64 + t0_4_11);
t0_5_5 += (1 - rt0_5_5) * t0_4_10;

int rt0_5_6 = (input[i] > (831+offset0));
int t0_5_6 = rt0_5_6 * (64 + t0_4_13);
t0_5_6 += (1 - rt0_5_6) * t0_4_12;

int rt0_5_7 = (input[i] > (959+offset0));
int t0_5_7 = rt0_5_7 * (64 + t0_4_15);
t0_5_7 += (1 - rt0_5_7) * t0_4_14;

int rt0_6_0 = (input[i] > (127+offset0));
int t0_6_0 = rt0_6_0 * (128 + t0_5_1);
t0_6_0 += (1 - rt0_6_0) * t0_5_0;

int rt0_6_1 = (input[i] > (383+offset0));
int t0_6_1 = rt0_6_1 * (128 + t0_5_3);
t0_6_1 += (1 - rt0_6_1) * t0_5_2;

int rt0_6_2 = (input[i] > (639+offset0));
int t0_6_2 = rt0_6_2 * (128 + t0_5_5);
t0_6_2 += (1 - rt0_6_2) * t0_5_4;

int rt0_6_3 = (input[i] > (895+offset0));
int t0_6_3 = rt0_6_3 * (128 + t0_5_7);
t0_6_3 += (1 - rt0_6_3) * t0_5_6;

int rt0_7_0 = (input[i] > (255+offset0));
int t0_7_0 = rt0_7_0 * (256 + t0_6_1);
t0_7_0 += (1 - rt0_7_0) * t0_6_0;

int rt0_7_1 = (input[i] > (767+offset0));
int t0_7_1 = rt0_7_1 * (256 + t0_6_3);
t0_7_1 += (1 - rt0_7_1) * t0_6_2;

int rt0_8_0 = (input[i] > (511+offset0));
int t0_8_0 = rt0_8_0 * (512 + t0_7_1);
t0_8_0 += (1 - rt0_8_0) * t0_7_0;

int rt1_0_0 = (input[i] > (1+offset1));
int t1_0_0 = rt1_0_0 * (2 + (input[i] > (2+offset1)));
t1_0_0 += (1 - rt1_0_0) * (input[i] > (0+offset1));

int rt1_0_1 = (input[i] > (5+offset1));
int t1_0_1 = rt1_0_1 * (2 + (input[i] > (6+offset1)));
t1_0_1 += (1 - rt1_0_1) * (input[i] > (4+offset1));

int rt1_0_2 = (input[i] > (9+offset1));
int t1_0_2 = rt1_0_2 * (2 + (input[i] > (10+offset1)));
t1_0_2 += (1 - rt1_0_2) * (input[i] > (8+offset1));

int rt1_0_3 = (input[i] > (13+offset1));
int t1_0_3 = rt1_0_3 * (2 + (input[i] > (14+offset1)));
t1_0_3 += (1 - rt1_0_3) * (input[i] > (12+offset1));

int rt1_0_4 = (input[i] > (17+offset1));
int t1_0_4 = rt1_0_4 * (2 + (input[i] > (18+offset1)));
t1_0_4 += (1 - rt1_0_4) * (input[i] > (16+offset1));

int rt1_0_5 = (input[i] > (21+offset1));
int t1_0_5 = rt1_0_5 * (2 + (input[i] > (22+offset1)));
t1_0_5 += (1 - rt1_0_5) * (input[i] > (20+offset1));

int rt1_0_6 = (input[i] > (25+offset1));
int t1_0_6 = rt1_0_6 * (2 + (input[i] > (26+offset1)));
t1_0_6 += (1 - rt1_0_6) * (input[i] > (24+offset1));

int rt1_0_7 = (input[i] > (29+offset1));
int t1_0_7 = rt1_0_7 * (2 + (input[i] > (30+offset1)));
t1_0_7 += (1 - rt1_0_7) * (input[i] > (28+offset1));

int rt1_0_8 = (input[i] > (33+offset1));
int t1_0_8 = rt1_0_8 * (2 + (input[i] > (34+offset1)));
t1_0_8 += (1 - rt1_0_8) * (input[i] > (32+offset1));

int rt1_0_9 = (input[i] > (37+offset1));
int t1_0_9 = rt1_0_9 * (2 + (input[i] > (38+offset1)));
t1_0_9 += (1 - rt1_0_9) * (input[i] > (36+offset1));

int rt1_0_10 = (input[i] > (41+offset1));
int t1_0_10 = rt1_0_10 * (2 + (input[i] > (42+offset1)));
t1_0_10 += (1 - rt1_0_10) * (input[i] > (40+offset1));

int rt1_0_11 = (input[i] > (45+offset1));
int t1_0_11 = rt1_0_11 * (2 + (input[i] > (46+offset1)));
t1_0_11 += (1 - rt1_0_11) * (input[i] > (44+offset1));

int rt1_0_12 = (input[i] > (49+offset1));
int t1_0_12 = rt1_0_12 * (2 + (input[i] > (50+offset1)));
t1_0_12 += (1 - rt1_0_12) * (input[i] > (48+offset1));

int rt1_0_13 = (input[i] > (53+offset1));
int t1_0_13 = rt1_0_13 * (2 + (input[i] > (54+offset1)));
t1_0_13 += (1 - rt1_0_13) * (input[i] > (52+offset1));

int rt1_0_14 = (input[i] > (57+offset1));
int t1_0_14 = rt1_0_14 * (2 + (input[i] > (58+offset1)));
t1_0_14 += (1 - rt1_0_14) * (input[i] > (56+offset1));

int rt1_0_15 = (input[i] > (61+offset1));
int t1_0_15 = rt1_0_15 * (2 + (input[i] > (62+offset1)));
t1_0_15 += (1 - rt1_0_15) * (input[i] > (60+offset1));

int rt1_0_16 = (input[i] > (65+offset1));
int t1_0_16 = rt1_0_16 * (2 + (input[i] > (66+offset1)));
t1_0_16 += (1 - rt1_0_16) * (input[i] > (64+offset1));

int rt1_0_17 = (input[i] > (69+offset1));
int t1_0_17 = rt1_0_17 * (2 + (input[i] > (70+offset1)));
t1_0_17 += (1 - rt1_0_17) * (input[i] > (68+offset1));

int rt1_0_18 = (input[i] > (73+offset1));
int t1_0_18 = rt1_0_18 * (2 + (input[i] > (74+offset1)));
t1_0_18 += (1 - rt1_0_18) * (input[i] > (72+offset1));

int rt1_0_19 = (input[i] > (77+offset1));
int t1_0_19 = rt1_0_19 * (2 + (input[i] > (78+offset1)));
t1_0_19 += (1 - rt1_0_19) * (input[i] > (76+offset1));

int rt1_0_20 = (input[i] > (81+offset1));
int t1_0_20 = rt1_0_20 * (2 + (input[i] > (82+offset1)));
t1_0_20 += (1 - rt1_0_20) * (input[i] > (80+offset1));

int rt1_0_21 = (input[i] > (85+offset1));
int t1_0_21 = rt1_0_21 * (2 + (input[i] > (86+offset1)));
t1_0_21 += (1 - rt1_0_21) * (input[i] > (84+offset1));

int rt1_0_22 = (input[i] > (89+offset1));
int t1_0_22 = rt1_0_22 * (2 + (input[i] > (90+offset1)));
t1_0_22 += (1 - rt1_0_22) * (input[i] > (88+offset1));

int rt1_0_23 = (input[i] > (93+offset1));
int t1_0_23 = rt1_0_23 * (2 + (input[i] > (94+offset1)));
t1_0_23 += (1 - rt1_0_23) * (input[i] > (92+offset1));

int rt1_0_24 = (input[i] > (97+offset1));
int t1_0_24 = rt1_0_24 * (2 + (input[i] > (98+offset1)));
t1_0_24 += (1 - rt1_0_24) * (input[i] > (96+offset1));

int rt1_0_25 = (input[i] > (101+offset1));
int t1_0_25 = rt1_0_25 * (2 + (input[i] > (102+offset1)));
t1_0_25 += (1 - rt1_0_25) * (input[i] > (100+offset1));

int rt1_0_26 = (input[i] > (105+offset1));
int t1_0_26 = rt1_0_26 * (2 + (input[i] > (106+offset1)));
t1_0_26 += (1 - rt1_0_26) * (input[i] > (104+offset1));

int rt1_0_27 = (input[i] > (109+offset1));
int t1_0_27 = rt1_0_27 * (2 + (input[i] > (110+offset1)));
t1_0_27 += (1 - rt1_0_27) * (input[i] > (108+offset1));

int rt1_0_28 = (input[i] > (113+offset1));
int t1_0_28 = rt1_0_28 * (2 + (input[i] > (114+offset1)));
t1_0_28 += (1 - rt1_0_28) * (input[i] > (112+offset1));

int rt1_0_29 = (input[i] > (117+offset1));
int t1_0_29 = rt1_0_29 * (2 + (input[i] > (118+offset1)));
t1_0_29 += (1 - rt1_0_29) * (input[i] > (116+offset1));

int rt1_0_30 = (input[i] > (121+offset1));
int t1_0_30 = rt1_0_30 * (2 + (input[i] > (122+offset1)));
t1_0_30 += (1 - rt1_0_30) * (input[i] > (120+offset1));

int rt1_0_31 = (input[i] > (125+offset1));
int t1_0_31 = rt1_0_31 * (2 + (input[i] > (126+offset1)));
t1_0_31 += (1 - rt1_0_31) * (input[i] > (124+offset1));

int rt1_0_32 = (input[i] > (129+offset1));
int t1_0_32 = rt1_0_32 * (2 + (input[i] > (130+offset1)));
t1_0_32 += (1 - rt1_0_32) * (input[i] > (128+offset1));

int rt1_0_33 = (input[i] > (133+offset1));
int t1_0_33 = rt1_0_33 * (2 + (input[i] > (134+offset1)));
t1_0_33 += (1 - rt1_0_33) * (input[i] > (132+offset1));

int rt1_0_34 = (input[i] > (137+offset1));
int t1_0_34 = rt1_0_34 * (2 + (input[i] > (138+offset1)));
t1_0_34 += (1 - rt1_0_34) * (input[i] > (136+offset1));

int rt1_0_35 = (input[i] > (141+offset1));
int t1_0_35 = rt1_0_35 * (2 + (input[i] > (142+offset1)));
t1_0_35 += (1 - rt1_0_35) * (input[i] > (140+offset1));

int rt1_0_36 = (input[i] > (145+offset1));
int t1_0_36 = rt1_0_36 * (2 + (input[i] > (146+offset1)));
t1_0_36 += (1 - rt1_0_36) * (input[i] > (144+offset1));

int rt1_0_37 = (input[i] > (149+offset1));
int t1_0_37 = rt1_0_37 * (2 + (input[i] > (150+offset1)));
t1_0_37 += (1 - rt1_0_37) * (input[i] > (148+offset1));

int rt1_0_38 = (input[i] > (153+offset1));
int t1_0_38 = rt1_0_38 * (2 + (input[i] > (154+offset1)));
t1_0_38 += (1 - rt1_0_38) * (input[i] > (152+offset1));

int rt1_0_39 = (input[i] > (157+offset1));
int t1_0_39 = rt1_0_39 * (2 + (input[i] > (158+offset1)));
t1_0_39 += (1 - rt1_0_39) * (input[i] > (156+offset1));

int rt1_0_40 = (input[i] > (161+offset1));
int t1_0_40 = rt1_0_40 * (2 + (input[i] > (162+offset1)));
t1_0_40 += (1 - rt1_0_40) * (input[i] > (160+offset1));

int rt1_0_41 = (input[i] > (165+offset1));
int t1_0_41 = rt1_0_41 * (2 + (input[i] > (166+offset1)));
t1_0_41 += (1 - rt1_0_41) * (input[i] > (164+offset1));

int rt1_0_42 = (input[i] > (169+offset1));
int t1_0_42 = rt1_0_42 * (2 + (input[i] > (170+offset1)));
t1_0_42 += (1 - rt1_0_42) * (input[i] > (168+offset1));

int rt1_0_43 = (input[i] > (173+offset1));
int t1_0_43 = rt1_0_43 * (2 + (input[i] > (174+offset1)));
t1_0_43 += (1 - rt1_0_43) * (input[i] > (172+offset1));

int rt1_0_44 = (input[i] > (177+offset1));
int t1_0_44 = rt1_0_44 * (2 + (input[i] > (178+offset1)));
t1_0_44 += (1 - rt1_0_44) * (input[i] > (176+offset1));

int rt1_0_45 = (input[i] > (181+offset1));
int t1_0_45 = rt1_0_45 * (2 + (input[i] > (182+offset1)));
t1_0_45 += (1 - rt1_0_45) * (input[i] > (180+offset1));

int rt1_0_46 = (input[i] > (185+offset1));
int t1_0_46 = rt1_0_46 * (2 + (input[i] > (186+offset1)));
t1_0_46 += (1 - rt1_0_46) * (input[i] > (184+offset1));

int rt1_0_47 = (input[i] > (189+offset1));
int t1_0_47 = rt1_0_47 * (2 + (input[i] > (190+offset1)));
t1_0_47 += (1 - rt1_0_47) * (input[i] > (188+offset1));

int rt1_0_48 = (input[i] > (193+offset1));
int t1_0_48 = rt1_0_48 * (2 + (input[i] > (194+offset1)));
t1_0_48 += (1 - rt1_0_48) * (input[i] > (192+offset1));

int rt1_0_49 = (input[i] > (197+offset1));
int t1_0_49 = rt1_0_49 * (2 + (input[i] > (198+offset1)));
t1_0_49 += (1 - rt1_0_49) * (input[i] > (196+offset1));

int rt1_0_50 = (input[i] > (201+offset1));
int t1_0_50 = rt1_0_50 * (2 + (input[i] > (202+offset1)));
t1_0_50 += (1 - rt1_0_50) * (input[i] > (200+offset1));

int rt1_0_51 = (input[i] > (205+offset1));
int t1_0_51 = rt1_0_51 * (2 + (input[i] > (206+offset1)));
t1_0_51 += (1 - rt1_0_51) * (input[i] > (204+offset1));

int rt1_0_52 = (input[i] > (209+offset1));
int t1_0_52 = rt1_0_52 * (2 + (input[i] > (210+offset1)));
t1_0_52 += (1 - rt1_0_52) * (input[i] > (208+offset1));

int rt1_0_53 = (input[i] > (213+offset1));
int t1_0_53 = rt1_0_53 * (2 + (input[i] > (214+offset1)));
t1_0_53 += (1 - rt1_0_53) * (input[i] > (212+offset1));

int rt1_0_54 = (input[i] > (217+offset1));
int t1_0_54 = rt1_0_54 * (2 + (input[i] > (218+offset1)));
t1_0_54 += (1 - rt1_0_54) * (input[i] > (216+offset1));

int rt1_0_55 = (input[i] > (221+offset1));
int t1_0_55 = rt1_0_55 * (2 + (input[i] > (222+offset1)));
t1_0_55 += (1 - rt1_0_55) * (input[i] > (220+offset1));

int rt1_0_56 = (input[i] > (225+offset1));
int t1_0_56 = rt1_0_56 * (2 + (input[i] > (226+offset1)));
t1_0_56 += (1 - rt1_0_56) * (input[i] > (224+offset1));

int rt1_0_57 = (input[i] > (229+offset1));
int t1_0_57 = rt1_0_57 * (2 + (input[i] > (230+offset1)));
t1_0_57 += (1 - rt1_0_57) * (input[i] > (228+offset1));

int rt1_0_58 = (input[i] > (233+offset1));
int t1_0_58 = rt1_0_58 * (2 + (input[i] > (234+offset1)));
t1_0_58 += (1 - rt1_0_58) * (input[i] > (232+offset1));

int rt1_0_59 = (input[i] > (237+offset1));
int t1_0_59 = rt1_0_59 * (2 + (input[i] > (238+offset1)));
t1_0_59 += (1 - rt1_0_59) * (input[i] > (236+offset1));

int rt1_0_60 = (input[i] > (241+offset1));
int t1_0_60 = rt1_0_60 * (2 + (input[i] > (242+offset1)));
t1_0_60 += (1 - rt1_0_60) * (input[i] > (240+offset1));

int rt1_0_61 = (input[i] > (245+offset1));
int t1_0_61 = rt1_0_61 * (2 + (input[i] > (246+offset1)));
t1_0_61 += (1 - rt1_0_61) * (input[i] > (244+offset1));

int rt1_0_62 = (input[i] > (249+offset1));
int t1_0_62 = rt1_0_62 * (2 + (input[i] > (250+offset1)));
t1_0_62 += (1 - rt1_0_62) * (input[i] > (248+offset1));

int rt1_0_63 = (input[i] > (253+offset1));
int t1_0_63 = rt1_0_63 * (2 + (input[i] > (254+offset1)));
t1_0_63 += (1 - rt1_0_63) * (input[i] > (252+offset1));

int rt1_0_64 = (input[i] > (257+offset1));
int t1_0_64 = rt1_0_64 * (2 + (input[i] > (258+offset1)));
t1_0_64 += (1 - rt1_0_64) * (input[i] > (256+offset1));

int rt1_0_65 = (input[i] > (261+offset1));
int t1_0_65 = rt1_0_65 * (2 + (input[i] > (262+offset1)));
t1_0_65 += (1 - rt1_0_65) * (input[i] > (260+offset1));

int rt1_0_66 = (input[i] > (265+offset1));
int t1_0_66 = rt1_0_66 * (2 + (input[i] > (266+offset1)));
t1_0_66 += (1 - rt1_0_66) * (input[i] > (264+offset1));

int rt1_0_67 = (input[i] > (269+offset1));
int t1_0_67 = rt1_0_67 * (2 + (input[i] > (270+offset1)));
t1_0_67 += (1 - rt1_0_67) * (input[i] > (268+offset1));

int rt1_0_68 = (input[i] > (273+offset1));
int t1_0_68 = rt1_0_68 * (2 + (input[i] > (274+offset1)));
t1_0_68 += (1 - rt1_0_68) * (input[i] > (272+offset1));

int rt1_0_69 = (input[i] > (277+offset1));
int t1_0_69 = rt1_0_69 * (2 + (input[i] > (278+offset1)));
t1_0_69 += (1 - rt1_0_69) * (input[i] > (276+offset1));

int rt1_0_70 = (input[i] > (281+offset1));
int t1_0_70 = rt1_0_70 * (2 + (input[i] > (282+offset1)));
t1_0_70 += (1 - rt1_0_70) * (input[i] > (280+offset1));

int rt1_0_71 = (input[i] > (285+offset1));
int t1_0_71 = rt1_0_71 * (2 + (input[i] > (286+offset1)));
t1_0_71 += (1 - rt1_0_71) * (input[i] > (284+offset1));

int rt1_0_72 = (input[i] > (289+offset1));
int t1_0_72 = rt1_0_72 * (2 + (input[i] > (290+offset1)));
t1_0_72 += (1 - rt1_0_72) * (input[i] > (288+offset1));

int rt1_0_73 = (input[i] > (293+offset1));
int t1_0_73 = rt1_0_73 * (2 + (input[i] > (294+offset1)));
t1_0_73 += (1 - rt1_0_73) * (input[i] > (292+offset1));

int rt1_0_74 = (input[i] > (297+offset1));
int t1_0_74 = rt1_0_74 * (2 + (input[i] > (298+offset1)));
t1_0_74 += (1 - rt1_0_74) * (input[i] > (296+offset1));

int rt1_0_75 = (input[i] > (301+offset1));
int t1_0_75 = rt1_0_75 * (2 + (input[i] > (302+offset1)));
t1_0_75 += (1 - rt1_0_75) * (input[i] > (300+offset1));

int rt1_0_76 = (input[i] > (305+offset1));
int t1_0_76 = rt1_0_76 * (2 + (input[i] > (306+offset1)));
t1_0_76 += (1 - rt1_0_76) * (input[i] > (304+offset1));

int rt1_0_77 = (input[i] > (309+offset1));
int t1_0_77 = rt1_0_77 * (2 + (input[i] > (310+offset1)));
t1_0_77 += (1 - rt1_0_77) * (input[i] > (308+offset1));

int rt1_0_78 = (input[i] > (313+offset1));
int t1_0_78 = rt1_0_78 * (2 + (input[i] > (314+offset1)));
t1_0_78 += (1 - rt1_0_78) * (input[i] > (312+offset1));

int rt1_0_79 = (input[i] > (317+offset1));
int t1_0_79 = rt1_0_79 * (2 + (input[i] > (318+offset1)));
t1_0_79 += (1 - rt1_0_79) * (input[i] > (316+offset1));

int rt1_0_80 = (input[i] > (321+offset1));
int t1_0_80 = rt1_0_80 * (2 + (input[i] > (322+offset1)));
t1_0_80 += (1 - rt1_0_80) * (input[i] > (320+offset1));

int rt1_0_81 = (input[i] > (325+offset1));
int t1_0_81 = rt1_0_81 * (2 + (input[i] > (326+offset1)));
t1_0_81 += (1 - rt1_0_81) * (input[i] > (324+offset1));

int rt1_0_82 = (input[i] > (329+offset1));
int t1_0_82 = rt1_0_82 * (2 + (input[i] > (330+offset1)));
t1_0_82 += (1 - rt1_0_82) * (input[i] > (328+offset1));

int rt1_0_83 = (input[i] > (333+offset1));
int t1_0_83 = rt1_0_83 * (2 + (input[i] > (334+offset1)));
t1_0_83 += (1 - rt1_0_83) * (input[i] > (332+offset1));

int rt1_0_84 = (input[i] > (337+offset1));
int t1_0_84 = rt1_0_84 * (2 + (input[i] > (338+offset1)));
t1_0_84 += (1 - rt1_0_84) * (input[i] > (336+offset1));

int rt1_0_85 = (input[i] > (341+offset1));
int t1_0_85 = rt1_0_85 * (2 + (input[i] > (342+offset1)));
t1_0_85 += (1 - rt1_0_85) * (input[i] > (340+offset1));

int rt1_0_86 = (input[i] > (345+offset1));
int t1_0_86 = rt1_0_86 * (2 + (input[i] > (346+offset1)));
t1_0_86 += (1 - rt1_0_86) * (input[i] > (344+offset1));

int rt1_0_87 = (input[i] > (349+offset1));
int t1_0_87 = rt1_0_87 * (2 + (input[i] > (350+offset1)));
t1_0_87 += (1 - rt1_0_87) * (input[i] > (348+offset1));

int rt1_0_88 = (input[i] > (353+offset1));
int t1_0_88 = rt1_0_88 * (2 + (input[i] > (354+offset1)));
t1_0_88 += (1 - rt1_0_88) * (input[i] > (352+offset1));

int rt1_0_89 = (input[i] > (357+offset1));
int t1_0_89 = rt1_0_89 * (2 + (input[i] > (358+offset1)));
t1_0_89 += (1 - rt1_0_89) * (input[i] > (356+offset1));

int rt1_0_90 = (input[i] > (361+offset1));
int t1_0_90 = rt1_0_90 * (2 + (input[i] > (362+offset1)));
t1_0_90 += (1 - rt1_0_90) * (input[i] > (360+offset1));

int rt1_0_91 = (input[i] > (365+offset1));
int t1_0_91 = rt1_0_91 * (2 + (input[i] > (366+offset1)));
t1_0_91 += (1 - rt1_0_91) * (input[i] > (364+offset1));

int rt1_0_92 = (input[i] > (369+offset1));
int t1_0_92 = rt1_0_92 * (2 + (input[i] > (370+offset1)));
t1_0_92 += (1 - rt1_0_92) * (input[i] > (368+offset1));

int rt1_0_93 = (input[i] > (373+offset1));
int t1_0_93 = rt1_0_93 * (2 + (input[i] > (374+offset1)));
t1_0_93 += (1 - rt1_0_93) * (input[i] > (372+offset1));

int rt1_0_94 = (input[i] > (377+offset1));
int t1_0_94 = rt1_0_94 * (2 + (input[i] > (378+offset1)));
t1_0_94 += (1 - rt1_0_94) * (input[i] > (376+offset1));

int rt1_0_95 = (input[i] > (381+offset1));
int t1_0_95 = rt1_0_95 * (2 + (input[i] > (382+offset1)));
t1_0_95 += (1 - rt1_0_95) * (input[i] > (380+offset1));

int rt1_0_96 = (input[i] > (385+offset1));
int t1_0_96 = rt1_0_96 * (2 + (input[i] > (386+offset1)));
t1_0_96 += (1 - rt1_0_96) * (input[i] > (384+offset1));

int rt1_0_97 = (input[i] > (389+offset1));
int t1_0_97 = rt1_0_97 * (2 + (input[i] > (390+offset1)));
t1_0_97 += (1 - rt1_0_97) * (input[i] > (388+offset1));

int rt1_0_98 = (input[i] > (393+offset1));
int t1_0_98 = rt1_0_98 * (2 + (input[i] > (394+offset1)));
t1_0_98 += (1 - rt1_0_98) * (input[i] > (392+offset1));

int rt1_0_99 = (input[i] > (397+offset1));
int t1_0_99 = rt1_0_99 * (2 + (input[i] > (398+offset1)));
t1_0_99 += (1 - rt1_0_99) * (input[i] > (396+offset1));

int rt1_0_100 = (input[i] > (401+offset1));
int t1_0_100 = rt1_0_100 * (2 + (input[i] > (402+offset1)));
t1_0_100 += (1 - rt1_0_100) * (input[i] > (400+offset1));

int rt1_0_101 = (input[i] > (405+offset1));
int t1_0_101 = rt1_0_101 * (2 + (input[i] > (406+offset1)));
t1_0_101 += (1 - rt1_0_101) * (input[i] > (404+offset1));

int rt1_0_102 = (input[i] > (409+offset1));
int t1_0_102 = rt1_0_102 * (2 + (input[i] > (410+offset1)));
t1_0_102 += (1 - rt1_0_102) * (input[i] > (408+offset1));

int rt1_0_103 = (input[i] > (413+offset1));
int t1_0_103 = rt1_0_103 * (2 + (input[i] > (414+offset1)));
t1_0_103 += (1 - rt1_0_103) * (input[i] > (412+offset1));

int rt1_0_104 = (input[i] > (417+offset1));
int t1_0_104 = rt1_0_104 * (2 + (input[i] > (418+offset1)));
t1_0_104 += (1 - rt1_0_104) * (input[i] > (416+offset1));

int rt1_0_105 = (input[i] > (421+offset1));
int t1_0_105 = rt1_0_105 * (2 + (input[i] > (422+offset1)));
t1_0_105 += (1 - rt1_0_105) * (input[i] > (420+offset1));

int rt1_0_106 = (input[i] > (425+offset1));
int t1_0_106 = rt1_0_106 * (2 + (input[i] > (426+offset1)));
t1_0_106 += (1 - rt1_0_106) * (input[i] > (424+offset1));

int rt1_0_107 = (input[i] > (429+offset1));
int t1_0_107 = rt1_0_107 * (2 + (input[i] > (430+offset1)));
t1_0_107 += (1 - rt1_0_107) * (input[i] > (428+offset1));

int rt1_0_108 = (input[i] > (433+offset1));
int t1_0_108 = rt1_0_108 * (2 + (input[i] > (434+offset1)));
t1_0_108 += (1 - rt1_0_108) * (input[i] > (432+offset1));

int rt1_0_109 = (input[i] > (437+offset1));
int t1_0_109 = rt1_0_109 * (2 + (input[i] > (438+offset1)));
t1_0_109 += (1 - rt1_0_109) * (input[i] > (436+offset1));

int rt1_0_110 = (input[i] > (441+offset1));
int t1_0_110 = rt1_0_110 * (2 + (input[i] > (442+offset1)));
t1_0_110 += (1 - rt1_0_110) * (input[i] > (440+offset1));

int rt1_0_111 = (input[i] > (445+offset1));
int t1_0_111 = rt1_0_111 * (2 + (input[i] > (446+offset1)));
t1_0_111 += (1 - rt1_0_111) * (input[i] > (444+offset1));

int rt1_0_112 = (input[i] > (449+offset1));
int t1_0_112 = rt1_0_112 * (2 + (input[i] > (450+offset1)));
t1_0_112 += (1 - rt1_0_112) * (input[i] > (448+offset1));

int rt1_0_113 = (input[i] > (453+offset1));
int t1_0_113 = rt1_0_113 * (2 + (input[i] > (454+offset1)));
t1_0_113 += (1 - rt1_0_113) * (input[i] > (452+offset1));

int rt1_0_114 = (input[i] > (457+offset1));
int t1_0_114 = rt1_0_114 * (2 + (input[i] > (458+offset1)));
t1_0_114 += (1 - rt1_0_114) * (input[i] > (456+offset1));

int rt1_0_115 = (input[i] > (461+offset1));
int t1_0_115 = rt1_0_115 * (2 + (input[i] > (462+offset1)));
t1_0_115 += (1 - rt1_0_115) * (input[i] > (460+offset1));

int rt1_0_116 = (input[i] > (465+offset1));
int t1_0_116 = rt1_0_116 * (2 + (input[i] > (466+offset1)));
t1_0_116 += (1 - rt1_0_116) * (input[i] > (464+offset1));

int rt1_0_117 = (input[i] > (469+offset1));
int t1_0_117 = rt1_0_117 * (2 + (input[i] > (470+offset1)));
t1_0_117 += (1 - rt1_0_117) * (input[i] > (468+offset1));

int rt1_0_118 = (input[i] > (473+offset1));
int t1_0_118 = rt1_0_118 * (2 + (input[i] > (474+offset1)));
t1_0_118 += (1 - rt1_0_118) * (input[i] > (472+offset1));

int rt1_0_119 = (input[i] > (477+offset1));
int t1_0_119 = rt1_0_119 * (2 + (input[i] > (478+offset1)));
t1_0_119 += (1 - rt1_0_119) * (input[i] > (476+offset1));

int rt1_0_120 = (input[i] > (481+offset1));
int t1_0_120 = rt1_0_120 * (2 + (input[i] > (482+offset1)));
t1_0_120 += (1 - rt1_0_120) * (input[i] > (480+offset1));

int rt1_0_121 = (input[i] > (485+offset1));
int t1_0_121 = rt1_0_121 * (2 + (input[i] > (486+offset1)));
t1_0_121 += (1 - rt1_0_121) * (input[i] > (484+offset1));

int rt1_0_122 = (input[i] > (489+offset1));
int t1_0_122 = rt1_0_122 * (2 + (input[i] > (490+offset1)));
t1_0_122 += (1 - rt1_0_122) * (input[i] > (488+offset1));

int rt1_0_123 = (input[i] > (493+offset1));
int t1_0_123 = rt1_0_123 * (2 + (input[i] > (494+offset1)));
t1_0_123 += (1 - rt1_0_123) * (input[i] > (492+offset1));

int rt1_0_124 = (input[i] > (497+offset1));
int t1_0_124 = rt1_0_124 * (2 + (input[i] > (498+offset1)));
t1_0_124 += (1 - rt1_0_124) * (input[i] > (496+offset1));

int rt1_0_125 = (input[i] > (501+offset1));
int t1_0_125 = rt1_0_125 * (2 + (input[i] > (502+offset1)));
t1_0_125 += (1 - rt1_0_125) * (input[i] > (500+offset1));

int rt1_0_126 = (input[i] > (505+offset1));
int t1_0_126 = rt1_0_126 * (2 + (input[i] > (506+offset1)));
t1_0_126 += (1 - rt1_0_126) * (input[i] > (504+offset1));

int rt1_0_127 = (input[i] > (509+offset1));
int t1_0_127 = rt1_0_127 * (2 + (input[i] > (510+offset1)));
t1_0_127 += (1 - rt1_0_127) * (input[i] > (508+offset1));

int rt1_0_128 = (input[i] > (513+offset1));
int t1_0_128 = rt1_0_128 * (2 + (input[i] > (514+offset1)));
t1_0_128 += (1 - rt1_0_128) * (input[i] > (512+offset1));

int rt1_0_129 = (input[i] > (517+offset1));
int t1_0_129 = rt1_0_129 * (2 + (input[i] > (518+offset1)));
t1_0_129 += (1 - rt1_0_129) * (input[i] > (516+offset1));

int rt1_0_130 = (input[i] > (521+offset1));
int t1_0_130 = rt1_0_130 * (2 + (input[i] > (522+offset1)));
t1_0_130 += (1 - rt1_0_130) * (input[i] > (520+offset1));

int rt1_0_131 = (input[i] > (525+offset1));
int t1_0_131 = rt1_0_131 * (2 + (input[i] > (526+offset1)));
t1_0_131 += (1 - rt1_0_131) * (input[i] > (524+offset1));

int rt1_0_132 = (input[i] > (529+offset1));
int t1_0_132 = rt1_0_132 * (2 + (input[i] > (530+offset1)));
t1_0_132 += (1 - rt1_0_132) * (input[i] > (528+offset1));

int rt1_0_133 = (input[i] > (533+offset1));
int t1_0_133 = rt1_0_133 * (2 + (input[i] > (534+offset1)));
t1_0_133 += (1 - rt1_0_133) * (input[i] > (532+offset1));

int rt1_0_134 = (input[i] > (537+offset1));
int t1_0_134 = rt1_0_134 * (2 + (input[i] > (538+offset1)));
t1_0_134 += (1 - rt1_0_134) * (input[i] > (536+offset1));

int rt1_0_135 = (input[i] > (541+offset1));
int t1_0_135 = rt1_0_135 * (2 + (input[i] > (542+offset1)));
t1_0_135 += (1 - rt1_0_135) * (input[i] > (540+offset1));

int rt1_0_136 = (input[i] > (545+offset1));
int t1_0_136 = rt1_0_136 * (2 + (input[i] > (546+offset1)));
t1_0_136 += (1 - rt1_0_136) * (input[i] > (544+offset1));

int rt1_0_137 = (input[i] > (549+offset1));
int t1_0_137 = rt1_0_137 * (2 + (input[i] > (550+offset1)));
t1_0_137 += (1 - rt1_0_137) * (input[i] > (548+offset1));

int rt1_0_138 = (input[i] > (553+offset1));
int t1_0_138 = rt1_0_138 * (2 + (input[i] > (554+offset1)));
t1_0_138 += (1 - rt1_0_138) * (input[i] > (552+offset1));

int rt1_0_139 = (input[i] > (557+offset1));
int t1_0_139 = rt1_0_139 * (2 + (input[i] > (558+offset1)));
t1_0_139 += (1 - rt1_0_139) * (input[i] > (556+offset1));

int rt1_0_140 = (input[i] > (561+offset1));
int t1_0_140 = rt1_0_140 * (2 + (input[i] > (562+offset1)));
t1_0_140 += (1 - rt1_0_140) * (input[i] > (560+offset1));

int rt1_0_141 = (input[i] > (565+offset1));
int t1_0_141 = rt1_0_141 * (2 + (input[i] > (566+offset1)));
t1_0_141 += (1 - rt1_0_141) * (input[i] > (564+offset1));

int rt1_0_142 = (input[i] > (569+offset1));
int t1_0_142 = rt1_0_142 * (2 + (input[i] > (570+offset1)));
t1_0_142 += (1 - rt1_0_142) * (input[i] > (568+offset1));

int rt1_0_143 = (input[i] > (573+offset1));
int t1_0_143 = rt1_0_143 * (2 + (input[i] > (574+offset1)));
t1_0_143 += (1 - rt1_0_143) * (input[i] > (572+offset1));

int rt1_0_144 = (input[i] > (577+offset1));
int t1_0_144 = rt1_0_144 * (2 + (input[i] > (578+offset1)));
t1_0_144 += (1 - rt1_0_144) * (input[i] > (576+offset1));

int rt1_0_145 = (input[i] > (581+offset1));
int t1_0_145 = rt1_0_145 * (2 + (input[i] > (582+offset1)));
t1_0_145 += (1 - rt1_0_145) * (input[i] > (580+offset1));

int rt1_0_146 = (input[i] > (585+offset1));
int t1_0_146 = rt1_0_146 * (2 + (input[i] > (586+offset1)));
t1_0_146 += (1 - rt1_0_146) * (input[i] > (584+offset1));

int rt1_0_147 = (input[i] > (589+offset1));
int t1_0_147 = rt1_0_147 * (2 + (input[i] > (590+offset1)));
t1_0_147 += (1 - rt1_0_147) * (input[i] > (588+offset1));

int rt1_0_148 = (input[i] > (593+offset1));
int t1_0_148 = rt1_0_148 * (2 + (input[i] > (594+offset1)));
t1_0_148 += (1 - rt1_0_148) * (input[i] > (592+offset1));

int rt1_0_149 = (input[i] > (597+offset1));
int t1_0_149 = rt1_0_149 * (2 + (input[i] > (598+offset1)));
t1_0_149 += (1 - rt1_0_149) * (input[i] > (596+offset1));

int rt1_0_150 = (input[i] > (601+offset1));
int t1_0_150 = rt1_0_150 * (2 + (input[i] > (602+offset1)));
t1_0_150 += (1 - rt1_0_150) * (input[i] > (600+offset1));

int rt1_0_151 = (input[i] > (605+offset1));
int t1_0_151 = rt1_0_151 * (2 + (input[i] > (606+offset1)));
t1_0_151 += (1 - rt1_0_151) * (input[i] > (604+offset1));

int rt1_0_152 = (input[i] > (609+offset1));
int t1_0_152 = rt1_0_152 * (2 + (input[i] > (610+offset1)));
t1_0_152 += (1 - rt1_0_152) * (input[i] > (608+offset1));

int rt1_0_153 = (input[i] > (613+offset1));
int t1_0_153 = rt1_0_153 * (2 + (input[i] > (614+offset1)));
t1_0_153 += (1 - rt1_0_153) * (input[i] > (612+offset1));

int rt1_0_154 = (input[i] > (617+offset1));
int t1_0_154 = rt1_0_154 * (2 + (input[i] > (618+offset1)));
t1_0_154 += (1 - rt1_0_154) * (input[i] > (616+offset1));

int rt1_0_155 = (input[i] > (621+offset1));
int t1_0_155 = rt1_0_155 * (2 + (input[i] > (622+offset1)));
t1_0_155 += (1 - rt1_0_155) * (input[i] > (620+offset1));

int rt1_0_156 = (input[i] > (625+offset1));
int t1_0_156 = rt1_0_156 * (2 + (input[i] > (626+offset1)));
t1_0_156 += (1 - rt1_0_156) * (input[i] > (624+offset1));

int rt1_0_157 = (input[i] > (629+offset1));
int t1_0_157 = rt1_0_157 * (2 + (input[i] > (630+offset1)));
t1_0_157 += (1 - rt1_0_157) * (input[i] > (628+offset1));

int rt1_0_158 = (input[i] > (633+offset1));
int t1_0_158 = rt1_0_158 * (2 + (input[i] > (634+offset1)));
t1_0_158 += (1 - rt1_0_158) * (input[i] > (632+offset1));

int rt1_0_159 = (input[i] > (637+offset1));
int t1_0_159 = rt1_0_159 * (2 + (input[i] > (638+offset1)));
t1_0_159 += (1 - rt1_0_159) * (input[i] > (636+offset1));

int rt1_0_160 = (input[i] > (641+offset1));
int t1_0_160 = rt1_0_160 * (2 + (input[i] > (642+offset1)));
t1_0_160 += (1 - rt1_0_160) * (input[i] > (640+offset1));

int rt1_0_161 = (input[i] > (645+offset1));
int t1_0_161 = rt1_0_161 * (2 + (input[i] > (646+offset1)));
t1_0_161 += (1 - rt1_0_161) * (input[i] > (644+offset1));

int rt1_0_162 = (input[i] > (649+offset1));
int t1_0_162 = rt1_0_162 * (2 + (input[i] > (650+offset1)));
t1_0_162 += (1 - rt1_0_162) * (input[i] > (648+offset1));

int rt1_0_163 = (input[i] > (653+offset1));
int t1_0_163 = rt1_0_163 * (2 + (input[i] > (654+offset1)));
t1_0_163 += (1 - rt1_0_163) * (input[i] > (652+offset1));

int rt1_0_164 = (input[i] > (657+offset1));
int t1_0_164 = rt1_0_164 * (2 + (input[i] > (658+offset1)));
t1_0_164 += (1 - rt1_0_164) * (input[i] > (656+offset1));

int rt1_0_165 = (input[i] > (661+offset1));
int t1_0_165 = rt1_0_165 * (2 + (input[i] > (662+offset1)));
t1_0_165 += (1 - rt1_0_165) * (input[i] > (660+offset1));

int rt1_0_166 = (input[i] > (665+offset1));
int t1_0_166 = rt1_0_166 * (2 + (input[i] > (666+offset1)));
t1_0_166 += (1 - rt1_0_166) * (input[i] > (664+offset1));

int rt1_0_167 = (input[i] > (669+offset1));
int t1_0_167 = rt1_0_167 * (2 + (input[i] > (670+offset1)));
t1_0_167 += (1 - rt1_0_167) * (input[i] > (668+offset1));

int rt1_0_168 = (input[i] > (673+offset1));
int t1_0_168 = rt1_0_168 * (2 + (input[i] > (674+offset1)));
t1_0_168 += (1 - rt1_0_168) * (input[i] > (672+offset1));

int rt1_0_169 = (input[i] > (677+offset1));
int t1_0_169 = rt1_0_169 * (2 + (input[i] > (678+offset1)));
t1_0_169 += (1 - rt1_0_169) * (input[i] > (676+offset1));

int rt1_0_170 = (input[i] > (681+offset1));
int t1_0_170 = rt1_0_170 * (2 + (input[i] > (682+offset1)));
t1_0_170 += (1 - rt1_0_170) * (input[i] > (680+offset1));

int rt1_0_171 = (input[i] > (685+offset1));
int t1_0_171 = rt1_0_171 * (2 + (input[i] > (686+offset1)));
t1_0_171 += (1 - rt1_0_171) * (input[i] > (684+offset1));

int rt1_0_172 = (input[i] > (689+offset1));
int t1_0_172 = rt1_0_172 * (2 + (input[i] > (690+offset1)));
t1_0_172 += (1 - rt1_0_172) * (input[i] > (688+offset1));

int rt1_0_173 = (input[i] > (693+offset1));
int t1_0_173 = rt1_0_173 * (2 + (input[i] > (694+offset1)));
t1_0_173 += (1 - rt1_0_173) * (input[i] > (692+offset1));

int rt1_0_174 = (input[i] > (697+offset1));
int t1_0_174 = rt1_0_174 * (2 + (input[i] > (698+offset1)));
t1_0_174 += (1 - rt1_0_174) * (input[i] > (696+offset1));

int rt1_0_175 = (input[i] > (701+offset1));
int t1_0_175 = rt1_0_175 * (2 + (input[i] > (702+offset1)));
t1_0_175 += (1 - rt1_0_175) * (input[i] > (700+offset1));

int rt1_0_176 = (input[i] > (705+offset1));
int t1_0_176 = rt1_0_176 * (2 + (input[i] > (706+offset1)));
t1_0_176 += (1 - rt1_0_176) * (input[i] > (704+offset1));

int rt1_0_177 = (input[i] > (709+offset1));
int t1_0_177 = rt1_0_177 * (2 + (input[i] > (710+offset1)));
t1_0_177 += (1 - rt1_0_177) * (input[i] > (708+offset1));

int rt1_0_178 = (input[i] > (713+offset1));
int t1_0_178 = rt1_0_178 * (2 + (input[i] > (714+offset1)));
t1_0_178 += (1 - rt1_0_178) * (input[i] > (712+offset1));

int rt1_0_179 = (input[i] > (717+offset1));
int t1_0_179 = rt1_0_179 * (2 + (input[i] > (718+offset1)));
t1_0_179 += (1 - rt1_0_179) * (input[i] > (716+offset1));

int rt1_0_180 = (input[i] > (721+offset1));
int t1_0_180 = rt1_0_180 * (2 + (input[i] > (722+offset1)));
t1_0_180 += (1 - rt1_0_180) * (input[i] > (720+offset1));

int rt1_0_181 = (input[i] > (725+offset1));
int t1_0_181 = rt1_0_181 * (2 + (input[i] > (726+offset1)));
t1_0_181 += (1 - rt1_0_181) * (input[i] > (724+offset1));

int rt1_0_182 = (input[i] > (729+offset1));
int t1_0_182 = rt1_0_182 * (2 + (input[i] > (730+offset1)));
t1_0_182 += (1 - rt1_0_182) * (input[i] > (728+offset1));

int rt1_0_183 = (input[i] > (733+offset1));
int t1_0_183 = rt1_0_183 * (2 + (input[i] > (734+offset1)));
t1_0_183 += (1 - rt1_0_183) * (input[i] > (732+offset1));

int rt1_0_184 = (input[i] > (737+offset1));
int t1_0_184 = rt1_0_184 * (2 + (input[i] > (738+offset1)));
t1_0_184 += (1 - rt1_0_184) * (input[i] > (736+offset1));

int rt1_0_185 = (input[i] > (741+offset1));
int t1_0_185 = rt1_0_185 * (2 + (input[i] > (742+offset1)));
t1_0_185 += (1 - rt1_0_185) * (input[i] > (740+offset1));

int rt1_0_186 = (input[i] > (745+offset1));
int t1_0_186 = rt1_0_186 * (2 + (input[i] > (746+offset1)));
t1_0_186 += (1 - rt1_0_186) * (input[i] > (744+offset1));

int rt1_0_187 = (input[i] > (749+offset1));
int t1_0_187 = rt1_0_187 * (2 + (input[i] > (750+offset1)));
t1_0_187 += (1 - rt1_0_187) * (input[i] > (748+offset1));

int rt1_0_188 = (input[i] > (753+offset1));
int t1_0_188 = rt1_0_188 * (2 + (input[i] > (754+offset1)));
t1_0_188 += (1 - rt1_0_188) * (input[i] > (752+offset1));

int rt1_0_189 = (input[i] > (757+offset1));
int t1_0_189 = rt1_0_189 * (2 + (input[i] > (758+offset1)));
t1_0_189 += (1 - rt1_0_189) * (input[i] > (756+offset1));

int rt1_0_190 = (input[i] > (761+offset1));
int t1_0_190 = rt1_0_190 * (2 + (input[i] > (762+offset1)));
t1_0_190 += (1 - rt1_0_190) * (input[i] > (760+offset1));

int rt1_0_191 = (input[i] > (765+offset1));
int t1_0_191 = rt1_0_191 * (2 + (input[i] > (766+offset1)));
t1_0_191 += (1 - rt1_0_191) * (input[i] > (764+offset1));

int rt1_0_192 = (input[i] > (769+offset1));
int t1_0_192 = rt1_0_192 * (2 + (input[i] > (770+offset1)));
t1_0_192 += (1 - rt1_0_192) * (input[i] > (768+offset1));

int rt1_0_193 = (input[i] > (773+offset1));
int t1_0_193 = rt1_0_193 * (2 + (input[i] > (774+offset1)));
t1_0_193 += (1 - rt1_0_193) * (input[i] > (772+offset1));

int rt1_0_194 = (input[i] > (777+offset1));
int t1_0_194 = rt1_0_194 * (2 + (input[i] > (778+offset1)));
t1_0_194 += (1 - rt1_0_194) * (input[i] > (776+offset1));

int rt1_0_195 = (input[i] > (781+offset1));
int t1_0_195 = rt1_0_195 * (2 + (input[i] > (782+offset1)));
t1_0_195 += (1 - rt1_0_195) * (input[i] > (780+offset1));

int rt1_0_196 = (input[i] > (785+offset1));
int t1_0_196 = rt1_0_196 * (2 + (input[i] > (786+offset1)));
t1_0_196 += (1 - rt1_0_196) * (input[i] > (784+offset1));

int rt1_0_197 = (input[i] > (789+offset1));
int t1_0_197 = rt1_0_197 * (2 + (input[i] > (790+offset1)));
t1_0_197 += (1 - rt1_0_197) * (input[i] > (788+offset1));

int rt1_0_198 = (input[i] > (793+offset1));
int t1_0_198 = rt1_0_198 * (2 + (input[i] > (794+offset1)));
t1_0_198 += (1 - rt1_0_198) * (input[i] > (792+offset1));

int rt1_0_199 = (input[i] > (797+offset1));
int t1_0_199 = rt1_0_199 * (2 + (input[i] > (798+offset1)));
t1_0_199 += (1 - rt1_0_199) * (input[i] > (796+offset1));

int rt1_0_200 = (input[i] > (801+offset1));
int t1_0_200 = rt1_0_200 * (2 + (input[i] > (802+offset1)));
t1_0_200 += (1 - rt1_0_200) * (input[i] > (800+offset1));

int rt1_0_201 = (input[i] > (805+offset1));
int t1_0_201 = rt1_0_201 * (2 + (input[i] > (806+offset1)));
t1_0_201 += (1 - rt1_0_201) * (input[i] > (804+offset1));

int rt1_0_202 = (input[i] > (809+offset1));
int t1_0_202 = rt1_0_202 * (2 + (input[i] > (810+offset1)));
t1_0_202 += (1 - rt1_0_202) * (input[i] > (808+offset1));

int rt1_0_203 = (input[i] > (813+offset1));
int t1_0_203 = rt1_0_203 * (2 + (input[i] > (814+offset1)));
t1_0_203 += (1 - rt1_0_203) * (input[i] > (812+offset1));

int rt1_0_204 = (input[i] > (817+offset1));
int t1_0_204 = rt1_0_204 * (2 + (input[i] > (818+offset1)));
t1_0_204 += (1 - rt1_0_204) * (input[i] > (816+offset1));

int rt1_0_205 = (input[i] > (821+offset1));
int t1_0_205 = rt1_0_205 * (2 + (input[i] > (822+offset1)));
t1_0_205 += (1 - rt1_0_205) * (input[i] > (820+offset1));

int rt1_0_206 = (input[i] > (825+offset1));
int t1_0_206 = rt1_0_206 * (2 + (input[i] > (826+offset1)));
t1_0_206 += (1 - rt1_0_206) * (input[i] > (824+offset1));

int rt1_0_207 = (input[i] > (829+offset1));
int t1_0_207 = rt1_0_207 * (2 + (input[i] > (830+offset1)));
t1_0_207 += (1 - rt1_0_207) * (input[i] > (828+offset1));

int rt1_0_208 = (input[i] > (833+offset1));
int t1_0_208 = rt1_0_208 * (2 + (input[i] > (834+offset1)));
t1_0_208 += (1 - rt1_0_208) * (input[i] > (832+offset1));

int rt1_0_209 = (input[i] > (837+offset1));
int t1_0_209 = rt1_0_209 * (2 + (input[i] > (838+offset1)));
t1_0_209 += (1 - rt1_0_209) * (input[i] > (836+offset1));

int rt1_0_210 = (input[i] > (841+offset1));
int t1_0_210 = rt1_0_210 * (2 + (input[i] > (842+offset1)));
t1_0_210 += (1 - rt1_0_210) * (input[i] > (840+offset1));

int rt1_0_211 = (input[i] > (845+offset1));
int t1_0_211 = rt1_0_211 * (2 + (input[i] > (846+offset1)));
t1_0_211 += (1 - rt1_0_211) * (input[i] > (844+offset1));

int rt1_0_212 = (input[i] > (849+offset1));
int t1_0_212 = rt1_0_212 * (2 + (input[i] > (850+offset1)));
t1_0_212 += (1 - rt1_0_212) * (input[i] > (848+offset1));

int rt1_0_213 = (input[i] > (853+offset1));
int t1_0_213 = rt1_0_213 * (2 + (input[i] > (854+offset1)));
t1_0_213 += (1 - rt1_0_213) * (input[i] > (852+offset1));

int rt1_0_214 = (input[i] > (857+offset1));
int t1_0_214 = rt1_0_214 * (2 + (input[i] > (858+offset1)));
t1_0_214 += (1 - rt1_0_214) * (input[i] > (856+offset1));

int rt1_0_215 = (input[i] > (861+offset1));
int t1_0_215 = rt1_0_215 * (2 + (input[i] > (862+offset1)));
t1_0_215 += (1 - rt1_0_215) * (input[i] > (860+offset1));

int rt1_0_216 = (input[i] > (865+offset1));
int t1_0_216 = rt1_0_216 * (2 + (input[i] > (866+offset1)));
t1_0_216 += (1 - rt1_0_216) * (input[i] > (864+offset1));

int rt1_0_217 = (input[i] > (869+offset1));
int t1_0_217 = rt1_0_217 * (2 + (input[i] > (870+offset1)));
t1_0_217 += (1 - rt1_0_217) * (input[i] > (868+offset1));

int rt1_0_218 = (input[i] > (873+offset1));
int t1_0_218 = rt1_0_218 * (2 + (input[i] > (874+offset1)));
t1_0_218 += (1 - rt1_0_218) * (input[i] > (872+offset1));

int rt1_0_219 = (input[i] > (877+offset1));
int t1_0_219 = rt1_0_219 * (2 + (input[i] > (878+offset1)));
t1_0_219 += (1 - rt1_0_219) * (input[i] > (876+offset1));

int rt1_0_220 = (input[i] > (881+offset1));
int t1_0_220 = rt1_0_220 * (2 + (input[i] > (882+offset1)));
t1_0_220 += (1 - rt1_0_220) * (input[i] > (880+offset1));

int rt1_0_221 = (input[i] > (885+offset1));
int t1_0_221 = rt1_0_221 * (2 + (input[i] > (886+offset1)));
t1_0_221 += (1 - rt1_0_221) * (input[i] > (884+offset1));

int rt1_0_222 = (input[i] > (889+offset1));
int t1_0_222 = rt1_0_222 * (2 + (input[i] > (890+offset1)));
t1_0_222 += (1 - rt1_0_222) * (input[i] > (888+offset1));

int rt1_0_223 = (input[i] > (893+offset1));
int t1_0_223 = rt1_0_223 * (2 + (input[i] > (894+offset1)));
t1_0_223 += (1 - rt1_0_223) * (input[i] > (892+offset1));

int rt1_0_224 = (input[i] > (897+offset1));
int t1_0_224 = rt1_0_224 * (2 + (input[i] > (898+offset1)));
t1_0_224 += (1 - rt1_0_224) * (input[i] > (896+offset1));

int rt1_0_225 = (input[i] > (901+offset1));
int t1_0_225 = rt1_0_225 * (2 + (input[i] > (902+offset1)));
t1_0_225 += (1 - rt1_0_225) * (input[i] > (900+offset1));

int rt1_0_226 = (input[i] > (905+offset1));
int t1_0_226 = rt1_0_226 * (2 + (input[i] > (906+offset1)));
t1_0_226 += (1 - rt1_0_226) * (input[i] > (904+offset1));

int rt1_0_227 = (input[i] > (909+offset1));
int t1_0_227 = rt1_0_227 * (2 + (input[i] > (910+offset1)));
t1_0_227 += (1 - rt1_0_227) * (input[i] > (908+offset1));

int rt1_0_228 = (input[i] > (913+offset1));
int t1_0_228 = rt1_0_228 * (2 + (input[i] > (914+offset1)));
t1_0_228 += (1 - rt1_0_228) * (input[i] > (912+offset1));

int rt1_0_229 = (input[i] > (917+offset1));
int t1_0_229 = rt1_0_229 * (2 + (input[i] > (918+offset1)));
t1_0_229 += (1 - rt1_0_229) * (input[i] > (916+offset1));

int rt1_0_230 = (input[i] > (921+offset1));
int t1_0_230 = rt1_0_230 * (2 + (input[i] > (922+offset1)));
t1_0_230 += (1 - rt1_0_230) * (input[i] > (920+offset1));

int rt1_0_231 = (input[i] > (925+offset1));
int t1_0_231 = rt1_0_231 * (2 + (input[i] > (926+offset1)));
t1_0_231 += (1 - rt1_0_231) * (input[i] > (924+offset1));

int rt1_0_232 = (input[i] > (929+offset1));
int t1_0_232 = rt1_0_232 * (2 + (input[i] > (930+offset1)));
t1_0_232 += (1 - rt1_0_232) * (input[i] > (928+offset1));

int rt1_0_233 = (input[i] > (933+offset1));
int t1_0_233 = rt1_0_233 * (2 + (input[i] > (934+offset1)));
t1_0_233 += (1 - rt1_0_233) * (input[i] > (932+offset1));

int rt1_0_234 = (input[i] > (937+offset1));
int t1_0_234 = rt1_0_234 * (2 + (input[i] > (938+offset1)));
t1_0_234 += (1 - rt1_0_234) * (input[i] > (936+offset1));

int rt1_0_235 = (input[i] > (941+offset1));
int t1_0_235 = rt1_0_235 * (2 + (input[i] > (942+offset1)));
t1_0_235 += (1 - rt1_0_235) * (input[i] > (940+offset1));

int rt1_0_236 = (input[i] > (945+offset1));
int t1_0_236 = rt1_0_236 * (2 + (input[i] > (946+offset1)));
t1_0_236 += (1 - rt1_0_236) * (input[i] > (944+offset1));

int rt1_0_237 = (input[i] > (949+offset1));
int t1_0_237 = rt1_0_237 * (2 + (input[i] > (950+offset1)));
t1_0_237 += (1 - rt1_0_237) * (input[i] > (948+offset1));

int rt1_0_238 = (input[i] > (953+offset1));
int t1_0_238 = rt1_0_238 * (2 + (input[i] > (954+offset1)));
t1_0_238 += (1 - rt1_0_238) * (input[i] > (952+offset1));

int rt1_0_239 = (input[i] > (957+offset1));
int t1_0_239 = rt1_0_239 * (2 + (input[i] > (958+offset1)));
t1_0_239 += (1 - rt1_0_239) * (input[i] > (956+offset1));

int rt1_0_240 = (input[i] > (961+offset1));
int t1_0_240 = rt1_0_240 * (2 + (input[i] > (962+offset1)));
t1_0_240 += (1 - rt1_0_240) * (input[i] > (960+offset1));

int rt1_0_241 = (input[i] > (965+offset1));
int t1_0_241 = rt1_0_241 * (2 + (input[i] > (966+offset1)));
t1_0_241 += (1 - rt1_0_241) * (input[i] > (964+offset1));

int rt1_0_242 = (input[i] > (969+offset1));
int t1_0_242 = rt1_0_242 * (2 + (input[i] > (970+offset1)));
t1_0_242 += (1 - rt1_0_242) * (input[i] > (968+offset1));

int rt1_0_243 = (input[i] > (973+offset1));
int t1_0_243 = rt1_0_243 * (2 + (input[i] > (974+offset1)));
t1_0_243 += (1 - rt1_0_243) * (input[i] > (972+offset1));

int rt1_0_244 = (input[i] > (977+offset1));
int t1_0_244 = rt1_0_244 * (2 + (input[i] > (978+offset1)));
t1_0_244 += (1 - rt1_0_244) * (input[i] > (976+offset1));

int rt1_0_245 = (input[i] > (981+offset1));
int t1_0_245 = rt1_0_245 * (2 + (input[i] > (982+offset1)));
t1_0_245 += (1 - rt1_0_245) * (input[i] > (980+offset1));

int rt1_0_246 = (input[i] > (985+offset1));
int t1_0_246 = rt1_0_246 * (2 + (input[i] > (986+offset1)));
t1_0_246 += (1 - rt1_0_246) * (input[i] > (984+offset1));

int rt1_0_247 = (input[i] > (989+offset1));
int t1_0_247 = rt1_0_247 * (2 + (input[i] > (990+offset1)));
t1_0_247 += (1 - rt1_0_247) * (input[i] > (988+offset1));

int rt1_0_248 = (input[i] > (993+offset1));
int t1_0_248 = rt1_0_248 * (2 + (input[i] > (994+offset1)));
t1_0_248 += (1 - rt1_0_248) * (input[i] > (992+offset1));

int rt1_0_249 = (input[i] > (997+offset1));
int t1_0_249 = rt1_0_249 * (2 + (input[i] > (998+offset1)));
t1_0_249 += (1 - rt1_0_249) * (input[i] > (996+offset1));

int rt1_0_250 = (input[i] > (1001+offset1));
int t1_0_250 = rt1_0_250 * (2 + (input[i] > (1002+offset1)));
t1_0_250 += (1 - rt1_0_250) * (input[i] > (1000+offset1));

int rt1_0_251 = (input[i] > (1005+offset1));
int t1_0_251 = rt1_0_251 * (2 + (input[i] > (1006+offset1)));
t1_0_251 += (1 - rt1_0_251) * (input[i] > (1004+offset1));

int rt1_0_252 = (input[i] > (1009+offset1));
int t1_0_252 = rt1_0_252 * (2 + (input[i] > (1010+offset1)));
t1_0_252 += (1 - rt1_0_252) * (input[i] > (1008+offset1));

int rt1_0_253 = (input[i] > (1013+offset1));
int t1_0_253 = rt1_0_253 * (2 + (input[i] > (1014+offset1)));
t1_0_253 += (1 - rt1_0_253) * (input[i] > (1012+offset1));

int rt1_0_254 = (input[i] > (1017+offset1));
int t1_0_254 = rt1_0_254 * (2 + (input[i] > (1018+offset1)));
t1_0_254 += (1 - rt1_0_254) * (input[i] > (1016+offset1));

int rt1_0_255 = (input[i] > (1021+offset1));
int t1_0_255 = rt1_0_255 * (2 + (input[i] > (1022+offset1)));
t1_0_255 += (1 - rt1_0_255) * (input[i] > (1020+offset1));

int rt1_1_0 = (input[i] > (3+offset1));
int t1_1_0 = rt1_1_0 * (4 + t1_0_1);
t1_1_0 += (1 - rt1_1_0) * t1_0_0;

int rt1_1_1 = (input[i] > (11+offset1));
int t1_1_1 = rt1_1_1 * (4 + t1_0_3);
t1_1_1 += (1 - rt1_1_1) * t1_0_2;

int rt1_1_2 = (input[i] > (19+offset1));
int t1_1_2 = rt1_1_2 * (4 + t1_0_5);
t1_1_2 += (1 - rt1_1_2) * t1_0_4;

int rt1_1_3 = (input[i] > (27+offset1));
int t1_1_3 = rt1_1_3 * (4 + t1_0_7);
t1_1_3 += (1 - rt1_1_3) * t1_0_6;

int rt1_1_4 = (input[i] > (35+offset1));
int t1_1_4 = rt1_1_4 * (4 + t1_0_9);
t1_1_4 += (1 - rt1_1_4) * t1_0_8;

int rt1_1_5 = (input[i] > (43+offset1));
int t1_1_5 = rt1_1_5 * (4 + t1_0_11);
t1_1_5 += (1 - rt1_1_5) * t1_0_10;

int rt1_1_6 = (input[i] > (51+offset1));
int t1_1_6 = rt1_1_6 * (4 + t1_0_13);
t1_1_6 += (1 - rt1_1_6) * t1_0_12;

int rt1_1_7 = (input[i] > (59+offset1));
int t1_1_7 = rt1_1_7 * (4 + t1_0_15);
t1_1_7 += (1 - rt1_1_7) * t1_0_14;

int rt1_1_8 = (input[i] > (67+offset1));
int t1_1_8 = rt1_1_8 * (4 + t1_0_17);
t1_1_8 += (1 - rt1_1_8) * t1_0_16;

int rt1_1_9 = (input[i] > (75+offset1));
int t1_1_9 = rt1_1_9 * (4 + t1_0_19);
t1_1_9 += (1 - rt1_1_9) * t1_0_18;

int rt1_1_10 = (input[i] > (83+offset1));
int t1_1_10 = rt1_1_10 * (4 + t1_0_21);
t1_1_10 += (1 - rt1_1_10) * t1_0_20;

int rt1_1_11 = (input[i] > (91+offset1));
int t1_1_11 = rt1_1_11 * (4 + t1_0_23);
t1_1_11 += (1 - rt1_1_11) * t1_0_22;

int rt1_1_12 = (input[i] > (99+offset1));
int t1_1_12 = rt1_1_12 * (4 + t1_0_25);
t1_1_12 += (1 - rt1_1_12) * t1_0_24;

int rt1_1_13 = (input[i] > (107+offset1));
int t1_1_13 = rt1_1_13 * (4 + t1_0_27);
t1_1_13 += (1 - rt1_1_13) * t1_0_26;

int rt1_1_14 = (input[i] > (115+offset1));
int t1_1_14 = rt1_1_14 * (4 + t1_0_29);
t1_1_14 += (1 - rt1_1_14) * t1_0_28;

int rt1_1_15 = (input[i] > (123+offset1));
int t1_1_15 = rt1_1_15 * (4 + t1_0_31);
t1_1_15 += (1 - rt1_1_15) * t1_0_30;

int rt1_1_16 = (input[i] > (131+offset1));
int t1_1_16 = rt1_1_16 * (4 + t1_0_33);
t1_1_16 += (1 - rt1_1_16) * t1_0_32;

int rt1_1_17 = (input[i] > (139+offset1));
int t1_1_17 = rt1_1_17 * (4 + t1_0_35);
t1_1_17 += (1 - rt1_1_17) * t1_0_34;

int rt1_1_18 = (input[i] > (147+offset1));
int t1_1_18 = rt1_1_18 * (4 + t1_0_37);
t1_1_18 += (1 - rt1_1_18) * t1_0_36;

int rt1_1_19 = (input[i] > (155+offset1));
int t1_1_19 = rt1_1_19 * (4 + t1_0_39);
t1_1_19 += (1 - rt1_1_19) * t1_0_38;

int rt1_1_20 = (input[i] > (163+offset1));
int t1_1_20 = rt1_1_20 * (4 + t1_0_41);
t1_1_20 += (1 - rt1_1_20) * t1_0_40;

int rt1_1_21 = (input[i] > (171+offset1));
int t1_1_21 = rt1_1_21 * (4 + t1_0_43);
t1_1_21 += (1 - rt1_1_21) * t1_0_42;

int rt1_1_22 = (input[i] > (179+offset1));
int t1_1_22 = rt1_1_22 * (4 + t1_0_45);
t1_1_22 += (1 - rt1_1_22) * t1_0_44;

int rt1_1_23 = (input[i] > (187+offset1));
int t1_1_23 = rt1_1_23 * (4 + t1_0_47);
t1_1_23 += (1 - rt1_1_23) * t1_0_46;

int rt1_1_24 = (input[i] > (195+offset1));
int t1_1_24 = rt1_1_24 * (4 + t1_0_49);
t1_1_24 += (1 - rt1_1_24) * t1_0_48;

int rt1_1_25 = (input[i] > (203+offset1));
int t1_1_25 = rt1_1_25 * (4 + t1_0_51);
t1_1_25 += (1 - rt1_1_25) * t1_0_50;

int rt1_1_26 = (input[i] > (211+offset1));
int t1_1_26 = rt1_1_26 * (4 + t1_0_53);
t1_1_26 += (1 - rt1_1_26) * t1_0_52;

int rt1_1_27 = (input[i] > (219+offset1));
int t1_1_27 = rt1_1_27 * (4 + t1_0_55);
t1_1_27 += (1 - rt1_1_27) * t1_0_54;

int rt1_1_28 = (input[i] > (227+offset1));
int t1_1_28 = rt1_1_28 * (4 + t1_0_57);
t1_1_28 += (1 - rt1_1_28) * t1_0_56;

int rt1_1_29 = (input[i] > (235+offset1));
int t1_1_29 = rt1_1_29 * (4 + t1_0_59);
t1_1_29 += (1 - rt1_1_29) * t1_0_58;

int rt1_1_30 = (input[i] > (243+offset1));
int t1_1_30 = rt1_1_30 * (4 + t1_0_61);
t1_1_30 += (1 - rt1_1_30) * t1_0_60;

int rt1_1_31 = (input[i] > (251+offset1));
int t1_1_31 = rt1_1_31 * (4 + t1_0_63);
t1_1_31 += (1 - rt1_1_31) * t1_0_62;

int rt1_1_32 = (input[i] > (259+offset1));
int t1_1_32 = rt1_1_32 * (4 + t1_0_65);
t1_1_32 += (1 - rt1_1_32) * t1_0_64;

int rt1_1_33 = (input[i] > (267+offset1));
int t1_1_33 = rt1_1_33 * (4 + t1_0_67);
t1_1_33 += (1 - rt1_1_33) * t1_0_66;

int rt1_1_34 = (input[i] > (275+offset1));
int t1_1_34 = rt1_1_34 * (4 + t1_0_69);
t1_1_34 += (1 - rt1_1_34) * t1_0_68;

int rt1_1_35 = (input[i] > (283+offset1));
int t1_1_35 = rt1_1_35 * (4 + t1_0_71);
t1_1_35 += (1 - rt1_1_35) * t1_0_70;

int rt1_1_36 = (input[i] > (291+offset1));
int t1_1_36 = rt1_1_36 * (4 + t1_0_73);
t1_1_36 += (1 - rt1_1_36) * t1_0_72;

int rt1_1_37 = (input[i] > (299+offset1));
int t1_1_37 = rt1_1_37 * (4 + t1_0_75);
t1_1_37 += (1 - rt1_1_37) * t1_0_74;

int rt1_1_38 = (input[i] > (307+offset1));
int t1_1_38 = rt1_1_38 * (4 + t1_0_77);
t1_1_38 += (1 - rt1_1_38) * t1_0_76;

int rt1_1_39 = (input[i] > (315+offset1));
int t1_1_39 = rt1_1_39 * (4 + t1_0_79);
t1_1_39 += (1 - rt1_1_39) * t1_0_78;

int rt1_1_40 = (input[i] > (323+offset1));
int t1_1_40 = rt1_1_40 * (4 + t1_0_81);
t1_1_40 += (1 - rt1_1_40) * t1_0_80;

int rt1_1_41 = (input[i] > (331+offset1));
int t1_1_41 = rt1_1_41 * (4 + t1_0_83);
t1_1_41 += (1 - rt1_1_41) * t1_0_82;

int rt1_1_42 = (input[i] > (339+offset1));
int t1_1_42 = rt1_1_42 * (4 + t1_0_85);
t1_1_42 += (1 - rt1_1_42) * t1_0_84;

int rt1_1_43 = (input[i] > (347+offset1));
int t1_1_43 = rt1_1_43 * (4 + t1_0_87);
t1_1_43 += (1 - rt1_1_43) * t1_0_86;

int rt1_1_44 = (input[i] > (355+offset1));
int t1_1_44 = rt1_1_44 * (4 + t1_0_89);
t1_1_44 += (1 - rt1_1_44) * t1_0_88;

int rt1_1_45 = (input[i] > (363+offset1));
int t1_1_45 = rt1_1_45 * (4 + t1_0_91);
t1_1_45 += (1 - rt1_1_45) * t1_0_90;

int rt1_1_46 = (input[i] > (371+offset1));
int t1_1_46 = rt1_1_46 * (4 + t1_0_93);
t1_1_46 += (1 - rt1_1_46) * t1_0_92;

int rt1_1_47 = (input[i] > (379+offset1));
int t1_1_47 = rt1_1_47 * (4 + t1_0_95);
t1_1_47 += (1 - rt1_1_47) * t1_0_94;

int rt1_1_48 = (input[i] > (387+offset1));
int t1_1_48 = rt1_1_48 * (4 + t1_0_97);
t1_1_48 += (1 - rt1_1_48) * t1_0_96;

int rt1_1_49 = (input[i] > (395+offset1));
int t1_1_49 = rt1_1_49 * (4 + t1_0_99);
t1_1_49 += (1 - rt1_1_49) * t1_0_98;

int rt1_1_50 = (input[i] > (403+offset1));
int t1_1_50 = rt1_1_50 * (4 + t1_0_101);
t1_1_50 += (1 - rt1_1_50) * t1_0_100;

int rt1_1_51 = (input[i] > (411+offset1));
int t1_1_51 = rt1_1_51 * (4 + t1_0_103);
t1_1_51 += (1 - rt1_1_51) * t1_0_102;

int rt1_1_52 = (input[i] > (419+offset1));
int t1_1_52 = rt1_1_52 * (4 + t1_0_105);
t1_1_52 += (1 - rt1_1_52) * t1_0_104;

int rt1_1_53 = (input[i] > (427+offset1));
int t1_1_53 = rt1_1_53 * (4 + t1_0_107);
t1_1_53 += (1 - rt1_1_53) * t1_0_106;

int rt1_1_54 = (input[i] > (435+offset1));
int t1_1_54 = rt1_1_54 * (4 + t1_0_109);
t1_1_54 += (1 - rt1_1_54) * t1_0_108;

int rt1_1_55 = (input[i] > (443+offset1));
int t1_1_55 = rt1_1_55 * (4 + t1_0_111);
t1_1_55 += (1 - rt1_1_55) * t1_0_110;

int rt1_1_56 = (input[i] > (451+offset1));
int t1_1_56 = rt1_1_56 * (4 + t1_0_113);
t1_1_56 += (1 - rt1_1_56) * t1_0_112;

int rt1_1_57 = (input[i] > (459+offset1));
int t1_1_57 = rt1_1_57 * (4 + t1_0_115);
t1_1_57 += (1 - rt1_1_57) * t1_0_114;

int rt1_1_58 = (input[i] > (467+offset1));
int t1_1_58 = rt1_1_58 * (4 + t1_0_117);
t1_1_58 += (1 - rt1_1_58) * t1_0_116;

int rt1_1_59 = (input[i] > (475+offset1));
int t1_1_59 = rt1_1_59 * (4 + t1_0_119);
t1_1_59 += (1 - rt1_1_59) * t1_0_118;

int rt1_1_60 = (input[i] > (483+offset1));
int t1_1_60 = rt1_1_60 * (4 + t1_0_121);
t1_1_60 += (1 - rt1_1_60) * t1_0_120;

int rt1_1_61 = (input[i] > (491+offset1));
int t1_1_61 = rt1_1_61 * (4 + t1_0_123);
t1_1_61 += (1 - rt1_1_61) * t1_0_122;

int rt1_1_62 = (input[i] > (499+offset1));
int t1_1_62 = rt1_1_62 * (4 + t1_0_125);
t1_1_62 += (1 - rt1_1_62) * t1_0_124;

int rt1_1_63 = (input[i] > (507+offset1));
int t1_1_63 = rt1_1_63 * (4 + t1_0_127);
t1_1_63 += (1 - rt1_1_63) * t1_0_126;

int rt1_1_64 = (input[i] > (515+offset1));
int t1_1_64 = rt1_1_64 * (4 + t1_0_129);
t1_1_64 += (1 - rt1_1_64) * t1_0_128;

int rt1_1_65 = (input[i] > (523+offset1));
int t1_1_65 = rt1_1_65 * (4 + t1_0_131);
t1_1_65 += (1 - rt1_1_65) * t1_0_130;

int rt1_1_66 = (input[i] > (531+offset1));
int t1_1_66 = rt1_1_66 * (4 + t1_0_133);
t1_1_66 += (1 - rt1_1_66) * t1_0_132;

int rt1_1_67 = (input[i] > (539+offset1));
int t1_1_67 = rt1_1_67 * (4 + t1_0_135);
t1_1_67 += (1 - rt1_1_67) * t1_0_134;

int rt1_1_68 = (input[i] > (547+offset1));
int t1_1_68 = rt1_1_68 * (4 + t1_0_137);
t1_1_68 += (1 - rt1_1_68) * t1_0_136;

int rt1_1_69 = (input[i] > (555+offset1));
int t1_1_69 = rt1_1_69 * (4 + t1_0_139);
t1_1_69 += (1 - rt1_1_69) * t1_0_138;

int rt1_1_70 = (input[i] > (563+offset1));
int t1_1_70 = rt1_1_70 * (4 + t1_0_141);
t1_1_70 += (1 - rt1_1_70) * t1_0_140;

int rt1_1_71 = (input[i] > (571+offset1));
int t1_1_71 = rt1_1_71 * (4 + t1_0_143);
t1_1_71 += (1 - rt1_1_71) * t1_0_142;

int rt1_1_72 = (input[i] > (579+offset1));
int t1_1_72 = rt1_1_72 * (4 + t1_0_145);
t1_1_72 += (1 - rt1_1_72) * t1_0_144;

int rt1_1_73 = (input[i] > (587+offset1));
int t1_1_73 = rt1_1_73 * (4 + t1_0_147);
t1_1_73 += (1 - rt1_1_73) * t1_0_146;

int rt1_1_74 = (input[i] > (595+offset1));
int t1_1_74 = rt1_1_74 * (4 + t1_0_149);
t1_1_74 += (1 - rt1_1_74) * t1_0_148;

int rt1_1_75 = (input[i] > (603+offset1));
int t1_1_75 = rt1_1_75 * (4 + t1_0_151);
t1_1_75 += (1 - rt1_1_75) * t1_0_150;

int rt1_1_76 = (input[i] > (611+offset1));
int t1_1_76 = rt1_1_76 * (4 + t1_0_153);
t1_1_76 += (1 - rt1_1_76) * t1_0_152;

int rt1_1_77 = (input[i] > (619+offset1));
int t1_1_77 = rt1_1_77 * (4 + t1_0_155);
t1_1_77 += (1 - rt1_1_77) * t1_0_154;

int rt1_1_78 = (input[i] > (627+offset1));
int t1_1_78 = rt1_1_78 * (4 + t1_0_157);
t1_1_78 += (1 - rt1_1_78) * t1_0_156;

int rt1_1_79 = (input[i] > (635+offset1));
int t1_1_79 = rt1_1_79 * (4 + t1_0_159);
t1_1_79 += (1 - rt1_1_79) * t1_0_158;

int rt1_1_80 = (input[i] > (643+offset1));
int t1_1_80 = rt1_1_80 * (4 + t1_0_161);
t1_1_80 += (1 - rt1_1_80) * t1_0_160;

int rt1_1_81 = (input[i] > (651+offset1));
int t1_1_81 = rt1_1_81 * (4 + t1_0_163);
t1_1_81 += (1 - rt1_1_81) * t1_0_162;

int rt1_1_82 = (input[i] > (659+offset1));
int t1_1_82 = rt1_1_82 * (4 + t1_0_165);
t1_1_82 += (1 - rt1_1_82) * t1_0_164;

int rt1_1_83 = (input[i] > (667+offset1));
int t1_1_83 = rt1_1_83 * (4 + t1_0_167);
t1_1_83 += (1 - rt1_1_83) * t1_0_166;

int rt1_1_84 = (input[i] > (675+offset1));
int t1_1_84 = rt1_1_84 * (4 + t1_0_169);
t1_1_84 += (1 - rt1_1_84) * t1_0_168;

int rt1_1_85 = (input[i] > (683+offset1));
int t1_1_85 = rt1_1_85 * (4 + t1_0_171);
t1_1_85 += (1 - rt1_1_85) * t1_0_170;

int rt1_1_86 = (input[i] > (691+offset1));
int t1_1_86 = rt1_1_86 * (4 + t1_0_173);
t1_1_86 += (1 - rt1_1_86) * t1_0_172;

int rt1_1_87 = (input[i] > (699+offset1));
int t1_1_87 = rt1_1_87 * (4 + t1_0_175);
t1_1_87 += (1 - rt1_1_87) * t1_0_174;

int rt1_1_88 = (input[i] > (707+offset1));
int t1_1_88 = rt1_1_88 * (4 + t1_0_177);
t1_1_88 += (1 - rt1_1_88) * t1_0_176;

int rt1_1_89 = (input[i] > (715+offset1));
int t1_1_89 = rt1_1_89 * (4 + t1_0_179);
t1_1_89 += (1 - rt1_1_89) * t1_0_178;

int rt1_1_90 = (input[i] > (723+offset1));
int t1_1_90 = rt1_1_90 * (4 + t1_0_181);
t1_1_90 += (1 - rt1_1_90) * t1_0_180;

int rt1_1_91 = (input[i] > (731+offset1));
int t1_1_91 = rt1_1_91 * (4 + t1_0_183);
t1_1_91 += (1 - rt1_1_91) * t1_0_182;

int rt1_1_92 = (input[i] > (739+offset1));
int t1_1_92 = rt1_1_92 * (4 + t1_0_185);
t1_1_92 += (1 - rt1_1_92) * t1_0_184;

int rt1_1_93 = (input[i] > (747+offset1));
int t1_1_93 = rt1_1_93 * (4 + t1_0_187);
t1_1_93 += (1 - rt1_1_93) * t1_0_186;

int rt1_1_94 = (input[i] > (755+offset1));
int t1_1_94 = rt1_1_94 * (4 + t1_0_189);
t1_1_94 += (1 - rt1_1_94) * t1_0_188;

int rt1_1_95 = (input[i] > (763+offset1));
int t1_1_95 = rt1_1_95 * (4 + t1_0_191);
t1_1_95 += (1 - rt1_1_95) * t1_0_190;

int rt1_1_96 = (input[i] > (771+offset1));
int t1_1_96 = rt1_1_96 * (4 + t1_0_193);
t1_1_96 += (1 - rt1_1_96) * t1_0_192;

int rt1_1_97 = (input[i] > (779+offset1));
int t1_1_97 = rt1_1_97 * (4 + t1_0_195);
t1_1_97 += (1 - rt1_1_97) * t1_0_194;

int rt1_1_98 = (input[i] > (787+offset1));
int t1_1_98 = rt1_1_98 * (4 + t1_0_197);
t1_1_98 += (1 - rt1_1_98) * t1_0_196;

int rt1_1_99 = (input[i] > (795+offset1));
int t1_1_99 = rt1_1_99 * (4 + t1_0_199);
t1_1_99 += (1 - rt1_1_99) * t1_0_198;

int rt1_1_100 = (input[i] > (803+offset1));
int t1_1_100 = rt1_1_100 * (4 + t1_0_201);
t1_1_100 += (1 - rt1_1_100) * t1_0_200;

int rt1_1_101 = (input[i] > (811+offset1));
int t1_1_101 = rt1_1_101 * (4 + t1_0_203);
t1_1_101 += (1 - rt1_1_101) * t1_0_202;

int rt1_1_102 = (input[i] > (819+offset1));
int t1_1_102 = rt1_1_102 * (4 + t1_0_205);
t1_1_102 += (1 - rt1_1_102) * t1_0_204;

int rt1_1_103 = (input[i] > (827+offset1));
int t1_1_103 = rt1_1_103 * (4 + t1_0_207);
t1_1_103 += (1 - rt1_1_103) * t1_0_206;

int rt1_1_104 = (input[i] > (835+offset1));
int t1_1_104 = rt1_1_104 * (4 + t1_0_209);
t1_1_104 += (1 - rt1_1_104) * t1_0_208;

int rt1_1_105 = (input[i] > (843+offset1));
int t1_1_105 = rt1_1_105 * (4 + t1_0_211);
t1_1_105 += (1 - rt1_1_105) * t1_0_210;

int rt1_1_106 = (input[i] > (851+offset1));
int t1_1_106 = rt1_1_106 * (4 + t1_0_213);
t1_1_106 += (1 - rt1_1_106) * t1_0_212;

int rt1_1_107 = (input[i] > (859+offset1));
int t1_1_107 = rt1_1_107 * (4 + t1_0_215);
t1_1_107 += (1 - rt1_1_107) * t1_0_214;

int rt1_1_108 = (input[i] > (867+offset1));
int t1_1_108 = rt1_1_108 * (4 + t1_0_217);
t1_1_108 += (1 - rt1_1_108) * t1_0_216;

int rt1_1_109 = (input[i] > (875+offset1));
int t1_1_109 = rt1_1_109 * (4 + t1_0_219);
t1_1_109 += (1 - rt1_1_109) * t1_0_218;

int rt1_1_110 = (input[i] > (883+offset1));
int t1_1_110 = rt1_1_110 * (4 + t1_0_221);
t1_1_110 += (1 - rt1_1_110) * t1_0_220;

int rt1_1_111 = (input[i] > (891+offset1));
int t1_1_111 = rt1_1_111 * (4 + t1_0_223);
t1_1_111 += (1 - rt1_1_111) * t1_0_222;

int rt1_1_112 = (input[i] > (899+offset1));
int t1_1_112 = rt1_1_112 * (4 + t1_0_225);
t1_1_112 += (1 - rt1_1_112) * t1_0_224;

int rt1_1_113 = (input[i] > (907+offset1));
int t1_1_113 = rt1_1_113 * (4 + t1_0_227);
t1_1_113 += (1 - rt1_1_113) * t1_0_226;

int rt1_1_114 = (input[i] > (915+offset1));
int t1_1_114 = rt1_1_114 * (4 + t1_0_229);
t1_1_114 += (1 - rt1_1_114) * t1_0_228;

int rt1_1_115 = (input[i] > (923+offset1));
int t1_1_115 = rt1_1_115 * (4 + t1_0_231);
t1_1_115 += (1 - rt1_1_115) * t1_0_230;

int rt1_1_116 = (input[i] > (931+offset1));
int t1_1_116 = rt1_1_116 * (4 + t1_0_233);
t1_1_116 += (1 - rt1_1_116) * t1_0_232;

int rt1_1_117 = (input[i] > (939+offset1));
int t1_1_117 = rt1_1_117 * (4 + t1_0_235);
t1_1_117 += (1 - rt1_1_117) * t1_0_234;

int rt1_1_118 = (input[i] > (947+offset1));
int t1_1_118 = rt1_1_118 * (4 + t1_0_237);
t1_1_118 += (1 - rt1_1_118) * t1_0_236;

int rt1_1_119 = (input[i] > (955+offset1));
int t1_1_119 = rt1_1_119 * (4 + t1_0_239);
t1_1_119 += (1 - rt1_1_119) * t1_0_238;

int rt1_1_120 = (input[i] > (963+offset1));
int t1_1_120 = rt1_1_120 * (4 + t1_0_241);
t1_1_120 += (1 - rt1_1_120) * t1_0_240;

int rt1_1_121 = (input[i] > (971+offset1));
int t1_1_121 = rt1_1_121 * (4 + t1_0_243);
t1_1_121 += (1 - rt1_1_121) * t1_0_242;

int rt1_1_122 = (input[i] > (979+offset1));
int t1_1_122 = rt1_1_122 * (4 + t1_0_245);
t1_1_122 += (1 - rt1_1_122) * t1_0_244;

int rt1_1_123 = (input[i] > (987+offset1));
int t1_1_123 = rt1_1_123 * (4 + t1_0_247);
t1_1_123 += (1 - rt1_1_123) * t1_0_246;

int rt1_1_124 = (input[i] > (995+offset1));
int t1_1_124 = rt1_1_124 * (4 + t1_0_249);
t1_1_124 += (1 - rt1_1_124) * t1_0_248;

int rt1_1_125 = (input[i] > (1003+offset1));
int t1_1_125 = rt1_1_125 * (4 + t1_0_251);
t1_1_125 += (1 - rt1_1_125) * t1_0_250;

int rt1_1_126 = (input[i] > (1011+offset1));
int t1_1_126 = rt1_1_126 * (4 + t1_0_253);
t1_1_126 += (1 - rt1_1_126) * t1_0_252;

int rt1_1_127 = (input[i] > (1019+offset1));
int t1_1_127 = rt1_1_127 * (4 + t1_0_255);
t1_1_127 += (1 - rt1_1_127) * t1_0_254;

int rt1_2_0 = (input[i] > (7+offset1));
int t1_2_0 = rt1_2_0 * (8 + t1_1_1);
t1_2_0 += (1 - rt1_2_0) * t1_1_0;

int rt1_2_1 = (input[i] > (23+offset1));
int t1_2_1 = rt1_2_1 * (8 + t1_1_3);
t1_2_1 += (1 - rt1_2_1) * t1_1_2;

int rt1_2_2 = (input[i] > (39+offset1));
int t1_2_2 = rt1_2_2 * (8 + t1_1_5);
t1_2_2 += (1 - rt1_2_2) * t1_1_4;

int rt1_2_3 = (input[i] > (55+offset1));
int t1_2_3 = rt1_2_3 * (8 + t1_1_7);
t1_2_3 += (1 - rt1_2_3) * t1_1_6;

int rt1_2_4 = (input[i] > (71+offset1));
int t1_2_4 = rt1_2_4 * (8 + t1_1_9);
t1_2_4 += (1 - rt1_2_4) * t1_1_8;

int rt1_2_5 = (input[i] > (87+offset1));
int t1_2_5 = rt1_2_5 * (8 + t1_1_11);
t1_2_5 += (1 - rt1_2_5) * t1_1_10;

int rt1_2_6 = (input[i] > (103+offset1));
int t1_2_6 = rt1_2_6 * (8 + t1_1_13);
t1_2_6 += (1 - rt1_2_6) * t1_1_12;

int rt1_2_7 = (input[i] > (119+offset1));
int t1_2_7 = rt1_2_7 * (8 + t1_1_15);
t1_2_7 += (1 - rt1_2_7) * t1_1_14;

int rt1_2_8 = (input[i] > (135+offset1));
int t1_2_8 = rt1_2_8 * (8 + t1_1_17);
t1_2_8 += (1 - rt1_2_8) * t1_1_16;

int rt1_2_9 = (input[i] > (151+offset1));
int t1_2_9 = rt1_2_9 * (8 + t1_1_19);
t1_2_9 += (1 - rt1_2_9) * t1_1_18;

int rt1_2_10 = (input[i] > (167+offset1));
int t1_2_10 = rt1_2_10 * (8 + t1_1_21);
t1_2_10 += (1 - rt1_2_10) * t1_1_20;

int rt1_2_11 = (input[i] > (183+offset1));
int t1_2_11 = rt1_2_11 * (8 + t1_1_23);
t1_2_11 += (1 - rt1_2_11) * t1_1_22;

int rt1_2_12 = (input[i] > (199+offset1));
int t1_2_12 = rt1_2_12 * (8 + t1_1_25);
t1_2_12 += (1 - rt1_2_12) * t1_1_24;

int rt1_2_13 = (input[i] > (215+offset1));
int t1_2_13 = rt1_2_13 * (8 + t1_1_27);
t1_2_13 += (1 - rt1_2_13) * t1_1_26;

int rt1_2_14 = (input[i] > (231+offset1));
int t1_2_14 = rt1_2_14 * (8 + t1_1_29);
t1_2_14 += (1 - rt1_2_14) * t1_1_28;

int rt1_2_15 = (input[i] > (247+offset1));
int t1_2_15 = rt1_2_15 * (8 + t1_1_31);
t1_2_15 += (1 - rt1_2_15) * t1_1_30;

int rt1_2_16 = (input[i] > (263+offset1));
int t1_2_16 = rt1_2_16 * (8 + t1_1_33);
t1_2_16 += (1 - rt1_2_16) * t1_1_32;

int rt1_2_17 = (input[i] > (279+offset1));
int t1_2_17 = rt1_2_17 * (8 + t1_1_35);
t1_2_17 += (1 - rt1_2_17) * t1_1_34;

int rt1_2_18 = (input[i] > (295+offset1));
int t1_2_18 = rt1_2_18 * (8 + t1_1_37);
t1_2_18 += (1 - rt1_2_18) * t1_1_36;

int rt1_2_19 = (input[i] > (311+offset1));
int t1_2_19 = rt1_2_19 * (8 + t1_1_39);
t1_2_19 += (1 - rt1_2_19) * t1_1_38;

int rt1_2_20 = (input[i] > (327+offset1));
int t1_2_20 = rt1_2_20 * (8 + t1_1_41);
t1_2_20 += (1 - rt1_2_20) * t1_1_40;

int rt1_2_21 = (input[i] > (343+offset1));
int t1_2_21 = rt1_2_21 * (8 + t1_1_43);
t1_2_21 += (1 - rt1_2_21) * t1_1_42;

int rt1_2_22 = (input[i] > (359+offset1));
int t1_2_22 = rt1_2_22 * (8 + t1_1_45);
t1_2_22 += (1 - rt1_2_22) * t1_1_44;

int rt1_2_23 = (input[i] > (375+offset1));
int t1_2_23 = rt1_2_23 * (8 + t1_1_47);
t1_2_23 += (1 - rt1_2_23) * t1_1_46;

int rt1_2_24 = (input[i] > (391+offset1));
int t1_2_24 = rt1_2_24 * (8 + t1_1_49);
t1_2_24 += (1 - rt1_2_24) * t1_1_48;

int rt1_2_25 = (input[i] > (407+offset1));
int t1_2_25 = rt1_2_25 * (8 + t1_1_51);
t1_2_25 += (1 - rt1_2_25) * t1_1_50;

int rt1_2_26 = (input[i] > (423+offset1));
int t1_2_26 = rt1_2_26 * (8 + t1_1_53);
t1_2_26 += (1 - rt1_2_26) * t1_1_52;

int rt1_2_27 = (input[i] > (439+offset1));
int t1_2_27 = rt1_2_27 * (8 + t1_1_55);
t1_2_27 += (1 - rt1_2_27) * t1_1_54;

int rt1_2_28 = (input[i] > (455+offset1));
int t1_2_28 = rt1_2_28 * (8 + t1_1_57);
t1_2_28 += (1 - rt1_2_28) * t1_1_56;

int rt1_2_29 = (input[i] > (471+offset1));
int t1_2_29 = rt1_2_29 * (8 + t1_1_59);
t1_2_29 += (1 - rt1_2_29) * t1_1_58;

int rt1_2_30 = (input[i] > (487+offset1));
int t1_2_30 = rt1_2_30 * (8 + t1_1_61);
t1_2_30 += (1 - rt1_2_30) * t1_1_60;

int rt1_2_31 = (input[i] > (503+offset1));
int t1_2_31 = rt1_2_31 * (8 + t1_1_63);
t1_2_31 += (1 - rt1_2_31) * t1_1_62;

int rt1_2_32 = (input[i] > (519+offset1));
int t1_2_32 = rt1_2_32 * (8 + t1_1_65);
t1_2_32 += (1 - rt1_2_32) * t1_1_64;

int rt1_2_33 = (input[i] > (535+offset1));
int t1_2_33 = rt1_2_33 * (8 + t1_1_67);
t1_2_33 += (1 - rt1_2_33) * t1_1_66;

int rt1_2_34 = (input[i] > (551+offset1));
int t1_2_34 = rt1_2_34 * (8 + t1_1_69);
t1_2_34 += (1 - rt1_2_34) * t1_1_68;

int rt1_2_35 = (input[i] > (567+offset1));
int t1_2_35 = rt1_2_35 * (8 + t1_1_71);
t1_2_35 += (1 - rt1_2_35) * t1_1_70;

int rt1_2_36 = (input[i] > (583+offset1));
int t1_2_36 = rt1_2_36 * (8 + t1_1_73);
t1_2_36 += (1 - rt1_2_36) * t1_1_72;

int rt1_2_37 = (input[i] > (599+offset1));
int t1_2_37 = rt1_2_37 * (8 + t1_1_75);
t1_2_37 += (1 - rt1_2_37) * t1_1_74;

int rt1_2_38 = (input[i] > (615+offset1));
int t1_2_38 = rt1_2_38 * (8 + t1_1_77);
t1_2_38 += (1 - rt1_2_38) * t1_1_76;

int rt1_2_39 = (input[i] > (631+offset1));
int t1_2_39 = rt1_2_39 * (8 + t1_1_79);
t1_2_39 += (1 - rt1_2_39) * t1_1_78;

int rt1_2_40 = (input[i] > (647+offset1));
int t1_2_40 = rt1_2_40 * (8 + t1_1_81);
t1_2_40 += (1 - rt1_2_40) * t1_1_80;

int rt1_2_41 = (input[i] > (663+offset1));
int t1_2_41 = rt1_2_41 * (8 + t1_1_83);
t1_2_41 += (1 - rt1_2_41) * t1_1_82;

int rt1_2_42 = (input[i] > (679+offset1));
int t1_2_42 = rt1_2_42 * (8 + t1_1_85);
t1_2_42 += (1 - rt1_2_42) * t1_1_84;

int rt1_2_43 = (input[i] > (695+offset1));
int t1_2_43 = rt1_2_43 * (8 + t1_1_87);
t1_2_43 += (1 - rt1_2_43) * t1_1_86;

int rt1_2_44 = (input[i] > (711+offset1));
int t1_2_44 = rt1_2_44 * (8 + t1_1_89);
t1_2_44 += (1 - rt1_2_44) * t1_1_88;

int rt1_2_45 = (input[i] > (727+offset1));
int t1_2_45 = rt1_2_45 * (8 + t1_1_91);
t1_2_45 += (1 - rt1_2_45) * t1_1_90;

int rt1_2_46 = (input[i] > (743+offset1));
int t1_2_46 = rt1_2_46 * (8 + t1_1_93);
t1_2_46 += (1 - rt1_2_46) * t1_1_92;

int rt1_2_47 = (input[i] > (759+offset1));
int t1_2_47 = rt1_2_47 * (8 + t1_1_95);
t1_2_47 += (1 - rt1_2_47) * t1_1_94;

int rt1_2_48 = (input[i] > (775+offset1));
int t1_2_48 = rt1_2_48 * (8 + t1_1_97);
t1_2_48 += (1 - rt1_2_48) * t1_1_96;

int rt1_2_49 = (input[i] > (791+offset1));
int t1_2_49 = rt1_2_49 * (8 + t1_1_99);
t1_2_49 += (1 - rt1_2_49) * t1_1_98;

int rt1_2_50 = (input[i] > (807+offset1));
int t1_2_50 = rt1_2_50 * (8 + t1_1_101);
t1_2_50 += (1 - rt1_2_50) * t1_1_100;

int rt1_2_51 = (input[i] > (823+offset1));
int t1_2_51 = rt1_2_51 * (8 + t1_1_103);
t1_2_51 += (1 - rt1_2_51) * t1_1_102;

int rt1_2_52 = (input[i] > (839+offset1));
int t1_2_52 = rt1_2_52 * (8 + t1_1_105);
t1_2_52 += (1 - rt1_2_52) * t1_1_104;

int rt1_2_53 = (input[i] > (855+offset1));
int t1_2_53 = rt1_2_53 * (8 + t1_1_107);
t1_2_53 += (1 - rt1_2_53) * t1_1_106;

int rt1_2_54 = (input[i] > (871+offset1));
int t1_2_54 = rt1_2_54 * (8 + t1_1_109);
t1_2_54 += (1 - rt1_2_54) * t1_1_108;

int rt1_2_55 = (input[i] > (887+offset1));
int t1_2_55 = rt1_2_55 * (8 + t1_1_111);
t1_2_55 += (1 - rt1_2_55) * t1_1_110;

int rt1_2_56 = (input[i] > (903+offset1));
int t1_2_56 = rt1_2_56 * (8 + t1_1_113);
t1_2_56 += (1 - rt1_2_56) * t1_1_112;

int rt1_2_57 = (input[i] > (919+offset1));
int t1_2_57 = rt1_2_57 * (8 + t1_1_115);
t1_2_57 += (1 - rt1_2_57) * t1_1_114;

int rt1_2_58 = (input[i] > (935+offset1));
int t1_2_58 = rt1_2_58 * (8 + t1_1_117);
t1_2_58 += (1 - rt1_2_58) * t1_1_116;

int rt1_2_59 = (input[i] > (951+offset1));
int t1_2_59 = rt1_2_59 * (8 + t1_1_119);
t1_2_59 += (1 - rt1_2_59) * t1_1_118;

int rt1_2_60 = (input[i] > (967+offset1));
int t1_2_60 = rt1_2_60 * (8 + t1_1_121);
t1_2_60 += (1 - rt1_2_60) * t1_1_120;

int rt1_2_61 = (input[i] > (983+offset1));
int t1_2_61 = rt1_2_61 * (8 + t1_1_123);
t1_2_61 += (1 - rt1_2_61) * t1_1_122;

int rt1_2_62 = (input[i] > (999+offset1));
int t1_2_62 = rt1_2_62 * (8 + t1_1_125);
t1_2_62 += (1 - rt1_2_62) * t1_1_124;

int rt1_2_63 = (input[i] > (1015+offset1));
int t1_2_63 = rt1_2_63 * (8 + t1_1_127);
t1_2_63 += (1 - rt1_2_63) * t1_1_126;

int rt1_3_0 = (input[i] > (15+offset1));
int t1_3_0 = rt1_3_0 * (16 + t1_2_1);
t1_3_0 += (1 - rt1_3_0) * t1_2_0;

int rt1_3_1 = (input[i] > (47+offset1));
int t1_3_1 = rt1_3_1 * (16 + t1_2_3);
t1_3_1 += (1 - rt1_3_1) * t1_2_2;

int rt1_3_2 = (input[i] > (79+offset1));
int t1_3_2 = rt1_3_2 * (16 + t1_2_5);
t1_3_2 += (1 - rt1_3_2) * t1_2_4;

int rt1_3_3 = (input[i] > (111+offset1));
int t1_3_3 = rt1_3_3 * (16 + t1_2_7);
t1_3_3 += (1 - rt1_3_3) * t1_2_6;

int rt1_3_4 = (input[i] > (143+offset1));
int t1_3_4 = rt1_3_4 * (16 + t1_2_9);
t1_3_4 += (1 - rt1_3_4) * t1_2_8;

int rt1_3_5 = (input[i] > (175+offset1));
int t1_3_5 = rt1_3_5 * (16 + t1_2_11);
t1_3_5 += (1 - rt1_3_5) * t1_2_10;

int rt1_3_6 = (input[i] > (207+offset1));
int t1_3_6 = rt1_3_6 * (16 + t1_2_13);
t1_3_6 += (1 - rt1_3_6) * t1_2_12;

int rt1_3_7 = (input[i] > (239+offset1));
int t1_3_7 = rt1_3_7 * (16 + t1_2_15);
t1_3_7 += (1 - rt1_3_7) * t1_2_14;

int rt1_3_8 = (input[i] > (271+offset1));
int t1_3_8 = rt1_3_8 * (16 + t1_2_17);
t1_3_8 += (1 - rt1_3_8) * t1_2_16;

int rt1_3_9 = (input[i] > (303+offset1));
int t1_3_9 = rt1_3_9 * (16 + t1_2_19);
t1_3_9 += (1 - rt1_3_9) * t1_2_18;

int rt1_3_10 = (input[i] > (335+offset1));
int t1_3_10 = rt1_3_10 * (16 + t1_2_21);
t1_3_10 += (1 - rt1_3_10) * t1_2_20;

int rt1_3_11 = (input[i] > (367+offset1));
int t1_3_11 = rt1_3_11 * (16 + t1_2_23);
t1_3_11 += (1 - rt1_3_11) * t1_2_22;

int rt1_3_12 = (input[i] > (399+offset1));
int t1_3_12 = rt1_3_12 * (16 + t1_2_25);
t1_3_12 += (1 - rt1_3_12) * t1_2_24;

int rt1_3_13 = (input[i] > (431+offset1));
int t1_3_13 = rt1_3_13 * (16 + t1_2_27);
t1_3_13 += (1 - rt1_3_13) * t1_2_26;

int rt1_3_14 = (input[i] > (463+offset1));
int t1_3_14 = rt1_3_14 * (16 + t1_2_29);
t1_3_14 += (1 - rt1_3_14) * t1_2_28;

int rt1_3_15 = (input[i] > (495+offset1));
int t1_3_15 = rt1_3_15 * (16 + t1_2_31);
t1_3_15 += (1 - rt1_3_15) * t1_2_30;

int rt1_3_16 = (input[i] > (527+offset1));
int t1_3_16 = rt1_3_16 * (16 + t1_2_33);
t1_3_16 += (1 - rt1_3_16) * t1_2_32;

int rt1_3_17 = (input[i] > (559+offset1));
int t1_3_17 = rt1_3_17 * (16 + t1_2_35);
t1_3_17 += (1 - rt1_3_17) * t1_2_34;

int rt1_3_18 = (input[i] > (591+offset1));
int t1_3_18 = rt1_3_18 * (16 + t1_2_37);
t1_3_18 += (1 - rt1_3_18) * t1_2_36;

int rt1_3_19 = (input[i] > (623+offset1));
int t1_3_19 = rt1_3_19 * (16 + t1_2_39);
t1_3_19 += (1 - rt1_3_19) * t1_2_38;

int rt1_3_20 = (input[i] > (655+offset1));
int t1_3_20 = rt1_3_20 * (16 + t1_2_41);
t1_3_20 += (1 - rt1_3_20) * t1_2_40;

int rt1_3_21 = (input[i] > (687+offset1));
int t1_3_21 = rt1_3_21 * (16 + t1_2_43);
t1_3_21 += (1 - rt1_3_21) * t1_2_42;

int rt1_3_22 = (input[i] > (719+offset1));
int t1_3_22 = rt1_3_22 * (16 + t1_2_45);
t1_3_22 += (1 - rt1_3_22) * t1_2_44;

int rt1_3_23 = (input[i] > (751+offset1));
int t1_3_23 = rt1_3_23 * (16 + t1_2_47);
t1_3_23 += (1 - rt1_3_23) * t1_2_46;

int rt1_3_24 = (input[i] > (783+offset1));
int t1_3_24 = rt1_3_24 * (16 + t1_2_49);
t1_3_24 += (1 - rt1_3_24) * t1_2_48;

int rt1_3_25 = (input[i] > (815+offset1));
int t1_3_25 = rt1_3_25 * (16 + t1_2_51);
t1_3_25 += (1 - rt1_3_25) * t1_2_50;

int rt1_3_26 = (input[i] > (847+offset1));
int t1_3_26 = rt1_3_26 * (16 + t1_2_53);
t1_3_26 += (1 - rt1_3_26) * t1_2_52;

int rt1_3_27 = (input[i] > (879+offset1));
int t1_3_27 = rt1_3_27 * (16 + t1_2_55);
t1_3_27 += (1 - rt1_3_27) * t1_2_54;

int rt1_3_28 = (input[i] > (911+offset1));
int t1_3_28 = rt1_3_28 * (16 + t1_2_57);
t1_3_28 += (1 - rt1_3_28) * t1_2_56;

int rt1_3_29 = (input[i] > (943+offset1));
int t1_3_29 = rt1_3_29 * (16 + t1_2_59);
t1_3_29 += (1 - rt1_3_29) * t1_2_58;

int rt1_3_30 = (input[i] > (975+offset1));
int t1_3_30 = rt1_3_30 * (16 + t1_2_61);
t1_3_30 += (1 - rt1_3_30) * t1_2_60;

int rt1_3_31 = (input[i] > (1007+offset1));
int t1_3_31 = rt1_3_31 * (16 + t1_2_63);
t1_3_31 += (1 - rt1_3_31) * t1_2_62;

int rt1_4_0 = (input[i] > (31+offset1));
int t1_4_0 = rt1_4_0 * (32 + t1_3_1);
t1_4_0 += (1 - rt1_4_0) * t1_3_0;

int rt1_4_1 = (input[i] > (95+offset1));
int t1_4_1 = rt1_4_1 * (32 + t1_3_3);
t1_4_1 += (1 - rt1_4_1) * t1_3_2;

int rt1_4_2 = (input[i] > (159+offset1));
int t1_4_2 = rt1_4_2 * (32 + t1_3_5);
t1_4_2 += (1 - rt1_4_2) * t1_3_4;

int rt1_4_3 = (input[i] > (223+offset1));
int t1_4_3 = rt1_4_3 * (32 + t1_3_7);
t1_4_3 += (1 - rt1_4_3) * t1_3_6;

int rt1_4_4 = (input[i] > (287+offset1));
int t1_4_4 = rt1_4_4 * (32 + t1_3_9);
t1_4_4 += (1 - rt1_4_4) * t1_3_8;

int rt1_4_5 = (input[i] > (351+offset1));
int t1_4_5 = rt1_4_5 * (32 + t1_3_11);
t1_4_5 += (1 - rt1_4_5) * t1_3_10;

int rt1_4_6 = (input[i] > (415+offset1));
int t1_4_6 = rt1_4_6 * (32 + t1_3_13);
t1_4_6 += (1 - rt1_4_6) * t1_3_12;

int rt1_4_7 = (input[i] > (479+offset1));
int t1_4_7 = rt1_4_7 * (32 + t1_3_15);
t1_4_7 += (1 - rt1_4_7) * t1_3_14;

int rt1_4_8 = (input[i] > (543+offset1));
int t1_4_8 = rt1_4_8 * (32 + t1_3_17);
t1_4_8 += (1 - rt1_4_8) * t1_3_16;

int rt1_4_9 = (input[i] > (607+offset1));
int t1_4_9 = rt1_4_9 * (32 + t1_3_19);
t1_4_9 += (1 - rt1_4_9) * t1_3_18;

int rt1_4_10 = (input[i] > (671+offset1));
int t1_4_10 = rt1_4_10 * (32 + t1_3_21);
t1_4_10 += (1 - rt1_4_10) * t1_3_20;

int rt1_4_11 = (input[i] > (735+offset1));
int t1_4_11 = rt1_4_11 * (32 + t1_3_23);
t1_4_11 += (1 - rt1_4_11) * t1_3_22;

int rt1_4_12 = (input[i] > (799+offset1));
int t1_4_12 = rt1_4_12 * (32 + t1_3_25);
t1_4_12 += (1 - rt1_4_12) * t1_3_24;

int rt1_4_13 = (input[i] > (863+offset1));
int t1_4_13 = rt1_4_13 * (32 + t1_3_27);
t1_4_13 += (1 - rt1_4_13) * t1_3_26;

int rt1_4_14 = (input[i] > (927+offset1));
int t1_4_14 = rt1_4_14 * (32 + t1_3_29);
t1_4_14 += (1 - rt1_4_14) * t1_3_28;

int rt1_4_15 = (input[i] > (991+offset1));
int t1_4_15 = rt1_4_15 * (32 + t1_3_31);
t1_4_15 += (1 - rt1_4_15) * t1_3_30;

int rt1_5_0 = (input[i] > (63+offset1));
int t1_5_0 = rt1_5_0 * (64 + t1_4_1);
t1_5_0 += (1 - rt1_5_0) * t1_4_0;

int rt1_5_1 = (input[i] > (191+offset1));
int t1_5_1 = rt1_5_1 * (64 + t1_4_3);
t1_5_1 += (1 - rt1_5_1) * t1_4_2;

int rt1_5_2 = (input[i] > (319+offset1));
int t1_5_2 = rt1_5_2 * (64 + t1_4_5);
t1_5_2 += (1 - rt1_5_2) * t1_4_4;

int rt1_5_3 = (input[i] > (447+offset1));
int t1_5_3 = rt1_5_3 * (64 + t1_4_7);
t1_5_3 += (1 - rt1_5_3) * t1_4_6;

int rt1_5_4 = (input[i] > (575+offset1));
int t1_5_4 = rt1_5_4 * (64 + t1_4_9);
t1_5_4 += (1 - rt1_5_4) * t1_4_8;

int rt1_5_5 = (input[i] > (703+offset1));
int t1_5_5 = rt1_5_5 * (64 + t1_4_11);
t1_5_5 += (1 - rt1_5_5) * t1_4_10;

int rt1_5_6 = (input[i] > (831+offset1));
int t1_5_6 = rt1_5_6 * (64 + t1_4_13);
t1_5_6 += (1 - rt1_5_6) * t1_4_12;

int rt1_5_7 = (input[i] > (959+offset1));
int t1_5_7 = rt1_5_7 * (64 + t1_4_15);
t1_5_7 += (1 - rt1_5_7) * t1_4_14;

int rt1_6_0 = (input[i] > (127+offset1));
int t1_6_0 = rt1_6_0 * (128 + t1_5_1);
t1_6_0 += (1 - rt1_6_0) * t1_5_0;

int rt1_6_1 = (input[i] > (383+offset1));
int t1_6_1 = rt1_6_1 * (128 + t1_5_3);
t1_6_1 += (1 - rt1_6_1) * t1_5_2;

int rt1_6_2 = (input[i] > (639+offset1));
int t1_6_2 = rt1_6_2 * (128 + t1_5_5);
t1_6_2 += (1 - rt1_6_2) * t1_5_4;

int rt1_6_3 = (input[i] > (895+offset1));
int t1_6_3 = rt1_6_3 * (128 + t1_5_7);
t1_6_3 += (1 - rt1_6_3) * t1_5_6;

int rt1_7_0 = (input[i] > (255+offset1));
int t1_7_0 = rt1_7_0 * (256 + t1_6_1);
t1_7_0 += (1 - rt1_7_0) * t1_6_0;

int rt1_7_1 = (input[i] > (767+offset1));
int t1_7_1 = rt1_7_1 * (256 + t1_6_3);
t1_7_1 += (1 - rt1_7_1) * t1_6_2;

int rt1_8_0 = (input[i] > (511+offset1));
int t1_8_0 = rt1_8_0 * (512 + t1_7_1);
t1_8_0 += (1 - rt1_8_0) * t1_7_0;

vR[i] = t0_8_0+ (t0_8_0 == t1_8_0);
	}
}

int main(int argc, char **argv)
{
	float elapsed_time;
	// set up device
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("[CUDA - Random Forest]: EXP %s\n", EXP_NAME);
	printf("[CUDA - Random Forest]: Using Device %d: %s\n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));

	// set up data size of vectors
	int nElem = N_ELEM;
	printf("[CUDA - Random Forest]: Vector Size %d\n", nElem);

	// malloc host memory
	size_t nBytes = nElem * sizeof(float);
	float *h_vR;
	h_vR = (float *)malloc(nBytes);
	float *h_input;
	h_input = (float *)malloc(nBytes);

	printf("[CUDA - Random Forest]: Start Reading Dataset\n");
	initData(h_input, nElem);
	printf("[CUDA - Random Forest]: Finished Reading Dataset\n");
	memset(h_vR, 0, nBytes);

	// malloc device global memory
	float *d_vR;
	CHECK(hipMalloc((float **)&d_vR, nBytes));
	float *d_input;
	CHECK(hipMalloc((float **)&d_input, nBytes));

	// transfer data from host to device
	CHECK(hipMemcpy(d_input, h_input, nBytes, hipMemcpyHostToDevice));

	// invoke kernel at host side
	int iLen = 512;
	dim3 block(iLen);
	dim3 grid((nElem + block.x - 1) / block.x);
	hipEvent_t start, stop;
	CHECK(hipEventCreate(&start));
	CHECK(hipEventCreate(&stop));

	// record start event
	CHECK(hipEventRecord(start, 0));
#if defined(_2TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, d_vR);
#elif defined(_3TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, d_vR);
#elif defined(_4TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, d_vR);
#else
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, d_vR);
#endif
	CHECK(hipEventRecord(stop, 0));
	CHECK(hipEventSynchronize(stop));
	hipDeviceSynchronize();

	// calculate elapsed time
	CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
	printf("[CUDA - Random Forest]: %s - execution time = %.6fms\n", EXP_NAME, elapsed_time);
	registerTime(elapsed_time);

	// copy kernel result back to host side
	CHECK(hipMemcpy(h_vR, d_vR, nBytes, hipMemcpyDeviceToHost));
	CHECK(hipGetLastError());

	//Verify the answers
	char flag = 1;
	for (int i = 0; i < nElem; i++)
	{
		if (i < 32)
		{
			printf("%.3f ", h_vR[i]);
		}
		if (h_input[i] != h_vR[i])
		{
			flag = 0;
			break;
		}
	}
	if (flag == 1)
	{
//		printf("Return correct");
	}
	else
	{
//		printf("Error!!");
	}
	printf("\n ");

	// free device memory
	hipFree(d_input);
	hipFree(d_vR);

	// free host memory
	free(h_input);
	free(h_vR);
	return (0);
}

void initData(float *ip, int size)
{
	for (int i = 0; i < size; i++)
	{
		ip[i] = (float)(0);
	}
	return;
}

void registerTime(float value)
{
	printf("[CUDA - Random Forest]: Time spent: %f.\n", value);
}
