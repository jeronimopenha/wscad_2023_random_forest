#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>

#define N_ELEM 50000000

#define WARP_SIZE 32

#define CHECK(call)                                                \
	{                                                              \
		const hipError_t error = call;                            \
		if (error != hipSuccess)                                  \
		{                                                          \
			fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
			fprintf(stderr, "code: %d, reason: %s\n", error,       \
					hipGetErrorString(error));                    \
		}                                                          \
	}

void initData(float *ip, int size);
void registerTime(float value);

#define _MOD 128
#define _2TREE
#define EXP_NAME "_RF_CALC_2TREE_7LV_1INPUT_50000000"

// RF_with_if

__global__ void RF_kernel(float *input, const int n, const int offset0, const int offset1, float *vR)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
int rt0_0_0 = (input[i] > (1+offset0));
int t0_0_0 = rt0_0_0 * (2 + (input[i] > (2+offset0)));
t0_0_0 += (1 - rt0_0_0) * (input[i] > (0+offset0));

int rt0_0_1 = (input[i] > (5+offset0));
int t0_0_1 = rt0_0_1 * (2 + (input[i] > (6+offset0)));
t0_0_1 += (1 - rt0_0_1) * (input[i] > (4+offset0));

int rt0_0_2 = (input[i] > (9+offset0));
int t0_0_2 = rt0_0_2 * (2 + (input[i] > (10+offset0)));
t0_0_2 += (1 - rt0_0_2) * (input[i] > (8+offset0));

int rt0_0_3 = (input[i] > (13+offset0));
int t0_0_3 = rt0_0_3 * (2 + (input[i] > (14+offset0)));
t0_0_3 += (1 - rt0_0_3) * (input[i] > (12+offset0));

int rt0_0_4 = (input[i] > (17+offset0));
int t0_0_4 = rt0_0_4 * (2 + (input[i] > (18+offset0)));
t0_0_4 += (1 - rt0_0_4) * (input[i] > (16+offset0));

int rt0_0_5 = (input[i] > (21+offset0));
int t0_0_5 = rt0_0_5 * (2 + (input[i] > (22+offset0)));
t0_0_5 += (1 - rt0_0_5) * (input[i] > (20+offset0));

int rt0_0_6 = (input[i] > (25+offset0));
int t0_0_6 = rt0_0_6 * (2 + (input[i] > (26+offset0)));
t0_0_6 += (1 - rt0_0_6) * (input[i] > (24+offset0));

int rt0_0_7 = (input[i] > (29+offset0));
int t0_0_7 = rt0_0_7 * (2 + (input[i] > (30+offset0)));
t0_0_7 += (1 - rt0_0_7) * (input[i] > (28+offset0));

int rt0_0_8 = (input[i] > (33+offset0));
int t0_0_8 = rt0_0_8 * (2 + (input[i] > (34+offset0)));
t0_0_8 += (1 - rt0_0_8) * (input[i] > (32+offset0));

int rt0_0_9 = (input[i] > (37+offset0));
int t0_0_9 = rt0_0_9 * (2 + (input[i] > (38+offset0)));
t0_0_9 += (1 - rt0_0_9) * (input[i] > (36+offset0));

int rt0_0_10 = (input[i] > (41+offset0));
int t0_0_10 = rt0_0_10 * (2 + (input[i] > (42+offset0)));
t0_0_10 += (1 - rt0_0_10) * (input[i] > (40+offset0));

int rt0_0_11 = (input[i] > (45+offset0));
int t0_0_11 = rt0_0_11 * (2 + (input[i] > (46+offset0)));
t0_0_11 += (1 - rt0_0_11) * (input[i] > (44+offset0));

int rt0_0_12 = (input[i] > (49+offset0));
int t0_0_12 = rt0_0_12 * (2 + (input[i] > (50+offset0)));
t0_0_12 += (1 - rt0_0_12) * (input[i] > (48+offset0));

int rt0_0_13 = (input[i] > (53+offset0));
int t0_0_13 = rt0_0_13 * (2 + (input[i] > (54+offset0)));
t0_0_13 += (1 - rt0_0_13) * (input[i] > (52+offset0));

int rt0_0_14 = (input[i] > (57+offset0));
int t0_0_14 = rt0_0_14 * (2 + (input[i] > (58+offset0)));
t0_0_14 += (1 - rt0_0_14) * (input[i] > (56+offset0));

int rt0_0_15 = (input[i] > (61+offset0));
int t0_0_15 = rt0_0_15 * (2 + (input[i] > (62+offset0)));
t0_0_15 += (1 - rt0_0_15) * (input[i] > (60+offset0));

int rt0_0_16 = (input[i] > (65+offset0));
int t0_0_16 = rt0_0_16 * (2 + (input[i] > (66+offset0)));
t0_0_16 += (1 - rt0_0_16) * (input[i] > (64+offset0));

int rt0_0_17 = (input[i] > (69+offset0));
int t0_0_17 = rt0_0_17 * (2 + (input[i] > (70+offset0)));
t0_0_17 += (1 - rt0_0_17) * (input[i] > (68+offset0));

int rt0_0_18 = (input[i] > (73+offset0));
int t0_0_18 = rt0_0_18 * (2 + (input[i] > (74+offset0)));
t0_0_18 += (1 - rt0_0_18) * (input[i] > (72+offset0));

int rt0_0_19 = (input[i] > (77+offset0));
int t0_0_19 = rt0_0_19 * (2 + (input[i] > (78+offset0)));
t0_0_19 += (1 - rt0_0_19) * (input[i] > (76+offset0));

int rt0_0_20 = (input[i] > (81+offset0));
int t0_0_20 = rt0_0_20 * (2 + (input[i] > (82+offset0)));
t0_0_20 += (1 - rt0_0_20) * (input[i] > (80+offset0));

int rt0_0_21 = (input[i] > (85+offset0));
int t0_0_21 = rt0_0_21 * (2 + (input[i] > (86+offset0)));
t0_0_21 += (1 - rt0_0_21) * (input[i] > (84+offset0));

int rt0_0_22 = (input[i] > (89+offset0));
int t0_0_22 = rt0_0_22 * (2 + (input[i] > (90+offset0)));
t0_0_22 += (1 - rt0_0_22) * (input[i] > (88+offset0));

int rt0_0_23 = (input[i] > (93+offset0));
int t0_0_23 = rt0_0_23 * (2 + (input[i] > (94+offset0)));
t0_0_23 += (1 - rt0_0_23) * (input[i] > (92+offset0));

int rt0_0_24 = (input[i] > (97+offset0));
int t0_0_24 = rt0_0_24 * (2 + (input[i] > (98+offset0)));
t0_0_24 += (1 - rt0_0_24) * (input[i] > (96+offset0));

int rt0_0_25 = (input[i] > (101+offset0));
int t0_0_25 = rt0_0_25 * (2 + (input[i] > (102+offset0)));
t0_0_25 += (1 - rt0_0_25) * (input[i] > (100+offset0));

int rt0_0_26 = (input[i] > (105+offset0));
int t0_0_26 = rt0_0_26 * (2 + (input[i] > (106+offset0)));
t0_0_26 += (1 - rt0_0_26) * (input[i] > (104+offset0));

int rt0_0_27 = (input[i] > (109+offset0));
int t0_0_27 = rt0_0_27 * (2 + (input[i] > (110+offset0)));
t0_0_27 += (1 - rt0_0_27) * (input[i] > (108+offset0));

int rt0_0_28 = (input[i] > (113+offset0));
int t0_0_28 = rt0_0_28 * (2 + (input[i] > (114+offset0)));
t0_0_28 += (1 - rt0_0_28) * (input[i] > (112+offset0));

int rt0_0_29 = (input[i] > (117+offset0));
int t0_0_29 = rt0_0_29 * (2 + (input[i] > (118+offset0)));
t0_0_29 += (1 - rt0_0_29) * (input[i] > (116+offset0));

int rt0_0_30 = (input[i] > (121+offset0));
int t0_0_30 = rt0_0_30 * (2 + (input[i] > (122+offset0)));
t0_0_30 += (1 - rt0_0_30) * (input[i] > (120+offset0));

int rt0_0_31 = (input[i] > (125+offset0));
int t0_0_31 = rt0_0_31 * (2 + (input[i] > (126+offset0)));
t0_0_31 += (1 - rt0_0_31) * (input[i] > (124+offset0));

int rt0_1_0 = (input[i] > (3+offset0));
int t0_1_0 = rt0_1_0 * (4 + t0_0_1);
t0_1_0 += (1 - rt0_1_0) * t0_0_0;

int rt0_1_1 = (input[i] > (11+offset0));
int t0_1_1 = rt0_1_1 * (4 + t0_0_3);
t0_1_1 += (1 - rt0_1_1) * t0_0_2;

int rt0_1_2 = (input[i] > (19+offset0));
int t0_1_2 = rt0_1_2 * (4 + t0_0_5);
t0_1_2 += (1 - rt0_1_2) * t0_0_4;

int rt0_1_3 = (input[i] > (27+offset0));
int t0_1_3 = rt0_1_3 * (4 + t0_0_7);
t0_1_3 += (1 - rt0_1_3) * t0_0_6;

int rt0_1_4 = (input[i] > (35+offset0));
int t0_1_4 = rt0_1_4 * (4 + t0_0_9);
t0_1_4 += (1 - rt0_1_4) * t0_0_8;

int rt0_1_5 = (input[i] > (43+offset0));
int t0_1_5 = rt0_1_5 * (4 + t0_0_11);
t0_1_5 += (1 - rt0_1_5) * t0_0_10;

int rt0_1_6 = (input[i] > (51+offset0));
int t0_1_6 = rt0_1_6 * (4 + t0_0_13);
t0_1_6 += (1 - rt0_1_6) * t0_0_12;

int rt0_1_7 = (input[i] > (59+offset0));
int t0_1_7 = rt0_1_7 * (4 + t0_0_15);
t0_1_7 += (1 - rt0_1_7) * t0_0_14;

int rt0_1_8 = (input[i] > (67+offset0));
int t0_1_8 = rt0_1_8 * (4 + t0_0_17);
t0_1_8 += (1 - rt0_1_8) * t0_0_16;

int rt0_1_9 = (input[i] > (75+offset0));
int t0_1_9 = rt0_1_9 * (4 + t0_0_19);
t0_1_9 += (1 - rt0_1_9) * t0_0_18;

int rt0_1_10 = (input[i] > (83+offset0));
int t0_1_10 = rt0_1_10 * (4 + t0_0_21);
t0_1_10 += (1 - rt0_1_10) * t0_0_20;

int rt0_1_11 = (input[i] > (91+offset0));
int t0_1_11 = rt0_1_11 * (4 + t0_0_23);
t0_1_11 += (1 - rt0_1_11) * t0_0_22;

int rt0_1_12 = (input[i] > (99+offset0));
int t0_1_12 = rt0_1_12 * (4 + t0_0_25);
t0_1_12 += (1 - rt0_1_12) * t0_0_24;

int rt0_1_13 = (input[i] > (107+offset0));
int t0_1_13 = rt0_1_13 * (4 + t0_0_27);
t0_1_13 += (1 - rt0_1_13) * t0_0_26;

int rt0_1_14 = (input[i] > (115+offset0));
int t0_1_14 = rt0_1_14 * (4 + t0_0_29);
t0_1_14 += (1 - rt0_1_14) * t0_0_28;

int rt0_1_15 = (input[i] > (123+offset0));
int t0_1_15 = rt0_1_15 * (4 + t0_0_31);
t0_1_15 += (1 - rt0_1_15) * t0_0_30;

int rt0_2_0 = (input[i] > (7+offset0));
int t0_2_0 = rt0_2_0 * (8 + t0_1_1);
t0_2_0 += (1 - rt0_2_0) * t0_1_0;

int rt0_2_1 = (input[i] > (23+offset0));
int t0_2_1 = rt0_2_1 * (8 + t0_1_3);
t0_2_1 += (1 - rt0_2_1) * t0_1_2;

int rt0_2_2 = (input[i] > (39+offset0));
int t0_2_2 = rt0_2_2 * (8 + t0_1_5);
t0_2_2 += (1 - rt0_2_2) * t0_1_4;

int rt0_2_3 = (input[i] > (55+offset0));
int t0_2_3 = rt0_2_3 * (8 + t0_1_7);
t0_2_3 += (1 - rt0_2_3) * t0_1_6;

int rt0_2_4 = (input[i] > (71+offset0));
int t0_2_4 = rt0_2_4 * (8 + t0_1_9);
t0_2_4 += (1 - rt0_2_4) * t0_1_8;

int rt0_2_5 = (input[i] > (87+offset0));
int t0_2_5 = rt0_2_5 * (8 + t0_1_11);
t0_2_5 += (1 - rt0_2_5) * t0_1_10;

int rt0_2_6 = (input[i] > (103+offset0));
int t0_2_6 = rt0_2_6 * (8 + t0_1_13);
t0_2_6 += (1 - rt0_2_6) * t0_1_12;

int rt0_2_7 = (input[i] > (119+offset0));
int t0_2_7 = rt0_2_7 * (8 + t0_1_15);
t0_2_7 += (1 - rt0_2_7) * t0_1_14;

int rt0_3_0 = (input[i] > (15+offset0));
int t0_3_0 = rt0_3_0 * (16 + t0_2_1);
t0_3_0 += (1 - rt0_3_0) * t0_2_0;

int rt0_3_1 = (input[i] > (47+offset0));
int t0_3_1 = rt0_3_1 * (16 + t0_2_3);
t0_3_1 += (1 - rt0_3_1) * t0_2_2;

int rt0_3_2 = (input[i] > (79+offset0));
int t0_3_2 = rt0_3_2 * (16 + t0_2_5);
t0_3_2 += (1 - rt0_3_2) * t0_2_4;

int rt0_3_3 = (input[i] > (111+offset0));
int t0_3_3 = rt0_3_3 * (16 + t0_2_7);
t0_3_3 += (1 - rt0_3_3) * t0_2_6;

int rt0_4_0 = (input[i] > (31+offset0));
int t0_4_0 = rt0_4_0 * (32 + t0_3_1);
t0_4_0 += (1 - rt0_4_0) * t0_3_0;

int rt0_4_1 = (input[i] > (95+offset0));
int t0_4_1 = rt0_4_1 * (32 + t0_3_3);
t0_4_1 += (1 - rt0_4_1) * t0_3_2;

int rt0_5_0 = (input[i] > (63+offset0));
int t0_5_0 = rt0_5_0 * (64 + t0_4_1);
t0_5_0 += (1 - rt0_5_0) * t0_4_0;

int rt1_0_0 = (input[i] > (1+offset1));
int t1_0_0 = rt1_0_0 * (2 + (input[i] > (2+offset1)));
t1_0_0 += (1 - rt1_0_0) * (input[i] > (0+offset1));

int rt1_0_1 = (input[i] > (5+offset1));
int t1_0_1 = rt1_0_1 * (2 + (input[i] > (6+offset1)));
t1_0_1 += (1 - rt1_0_1) * (input[i] > (4+offset1));

int rt1_0_2 = (input[i] > (9+offset1));
int t1_0_2 = rt1_0_2 * (2 + (input[i] > (10+offset1)));
t1_0_2 += (1 - rt1_0_2) * (input[i] > (8+offset1));

int rt1_0_3 = (input[i] > (13+offset1));
int t1_0_3 = rt1_0_3 * (2 + (input[i] > (14+offset1)));
t1_0_3 += (1 - rt1_0_3) * (input[i] > (12+offset1));

int rt1_0_4 = (input[i] > (17+offset1));
int t1_0_4 = rt1_0_4 * (2 + (input[i] > (18+offset1)));
t1_0_4 += (1 - rt1_0_4) * (input[i] > (16+offset1));

int rt1_0_5 = (input[i] > (21+offset1));
int t1_0_5 = rt1_0_5 * (2 + (input[i] > (22+offset1)));
t1_0_5 += (1 - rt1_0_5) * (input[i] > (20+offset1));

int rt1_0_6 = (input[i] > (25+offset1));
int t1_0_6 = rt1_0_6 * (2 + (input[i] > (26+offset1)));
t1_0_6 += (1 - rt1_0_6) * (input[i] > (24+offset1));

int rt1_0_7 = (input[i] > (29+offset1));
int t1_0_7 = rt1_0_7 * (2 + (input[i] > (30+offset1)));
t1_0_7 += (1 - rt1_0_7) * (input[i] > (28+offset1));

int rt1_0_8 = (input[i] > (33+offset1));
int t1_0_8 = rt1_0_8 * (2 + (input[i] > (34+offset1)));
t1_0_8 += (1 - rt1_0_8) * (input[i] > (32+offset1));

int rt1_0_9 = (input[i] > (37+offset1));
int t1_0_9 = rt1_0_9 * (2 + (input[i] > (38+offset1)));
t1_0_9 += (1 - rt1_0_9) * (input[i] > (36+offset1));

int rt1_0_10 = (input[i] > (41+offset1));
int t1_0_10 = rt1_0_10 * (2 + (input[i] > (42+offset1)));
t1_0_10 += (1 - rt1_0_10) * (input[i] > (40+offset1));

int rt1_0_11 = (input[i] > (45+offset1));
int t1_0_11 = rt1_0_11 * (2 + (input[i] > (46+offset1)));
t1_0_11 += (1 - rt1_0_11) * (input[i] > (44+offset1));

int rt1_0_12 = (input[i] > (49+offset1));
int t1_0_12 = rt1_0_12 * (2 + (input[i] > (50+offset1)));
t1_0_12 += (1 - rt1_0_12) * (input[i] > (48+offset1));

int rt1_0_13 = (input[i] > (53+offset1));
int t1_0_13 = rt1_0_13 * (2 + (input[i] > (54+offset1)));
t1_0_13 += (1 - rt1_0_13) * (input[i] > (52+offset1));

int rt1_0_14 = (input[i] > (57+offset1));
int t1_0_14 = rt1_0_14 * (2 + (input[i] > (58+offset1)));
t1_0_14 += (1 - rt1_0_14) * (input[i] > (56+offset1));

int rt1_0_15 = (input[i] > (61+offset1));
int t1_0_15 = rt1_0_15 * (2 + (input[i] > (62+offset1)));
t1_0_15 += (1 - rt1_0_15) * (input[i] > (60+offset1));

int rt1_0_16 = (input[i] > (65+offset1));
int t1_0_16 = rt1_0_16 * (2 + (input[i] > (66+offset1)));
t1_0_16 += (1 - rt1_0_16) * (input[i] > (64+offset1));

int rt1_0_17 = (input[i] > (69+offset1));
int t1_0_17 = rt1_0_17 * (2 + (input[i] > (70+offset1)));
t1_0_17 += (1 - rt1_0_17) * (input[i] > (68+offset1));

int rt1_0_18 = (input[i] > (73+offset1));
int t1_0_18 = rt1_0_18 * (2 + (input[i] > (74+offset1)));
t1_0_18 += (1 - rt1_0_18) * (input[i] > (72+offset1));

int rt1_0_19 = (input[i] > (77+offset1));
int t1_0_19 = rt1_0_19 * (2 + (input[i] > (78+offset1)));
t1_0_19 += (1 - rt1_0_19) * (input[i] > (76+offset1));

int rt1_0_20 = (input[i] > (81+offset1));
int t1_0_20 = rt1_0_20 * (2 + (input[i] > (82+offset1)));
t1_0_20 += (1 - rt1_0_20) * (input[i] > (80+offset1));

int rt1_0_21 = (input[i] > (85+offset1));
int t1_0_21 = rt1_0_21 * (2 + (input[i] > (86+offset1)));
t1_0_21 += (1 - rt1_0_21) * (input[i] > (84+offset1));

int rt1_0_22 = (input[i] > (89+offset1));
int t1_0_22 = rt1_0_22 * (2 + (input[i] > (90+offset1)));
t1_0_22 += (1 - rt1_0_22) * (input[i] > (88+offset1));

int rt1_0_23 = (input[i] > (93+offset1));
int t1_0_23 = rt1_0_23 * (2 + (input[i] > (94+offset1)));
t1_0_23 += (1 - rt1_0_23) * (input[i] > (92+offset1));

int rt1_0_24 = (input[i] > (97+offset1));
int t1_0_24 = rt1_0_24 * (2 + (input[i] > (98+offset1)));
t1_0_24 += (1 - rt1_0_24) * (input[i] > (96+offset1));

int rt1_0_25 = (input[i] > (101+offset1));
int t1_0_25 = rt1_0_25 * (2 + (input[i] > (102+offset1)));
t1_0_25 += (1 - rt1_0_25) * (input[i] > (100+offset1));

int rt1_0_26 = (input[i] > (105+offset1));
int t1_0_26 = rt1_0_26 * (2 + (input[i] > (106+offset1)));
t1_0_26 += (1 - rt1_0_26) * (input[i] > (104+offset1));

int rt1_0_27 = (input[i] > (109+offset1));
int t1_0_27 = rt1_0_27 * (2 + (input[i] > (110+offset1)));
t1_0_27 += (1 - rt1_0_27) * (input[i] > (108+offset1));

int rt1_0_28 = (input[i] > (113+offset1));
int t1_0_28 = rt1_0_28 * (2 + (input[i] > (114+offset1)));
t1_0_28 += (1 - rt1_0_28) * (input[i] > (112+offset1));

int rt1_0_29 = (input[i] > (117+offset1));
int t1_0_29 = rt1_0_29 * (2 + (input[i] > (118+offset1)));
t1_0_29 += (1 - rt1_0_29) * (input[i] > (116+offset1));

int rt1_0_30 = (input[i] > (121+offset1));
int t1_0_30 = rt1_0_30 * (2 + (input[i] > (122+offset1)));
t1_0_30 += (1 - rt1_0_30) * (input[i] > (120+offset1));

int rt1_0_31 = (input[i] > (125+offset1));
int t1_0_31 = rt1_0_31 * (2 + (input[i] > (126+offset1)));
t1_0_31 += (1 - rt1_0_31) * (input[i] > (124+offset1));

int rt1_1_0 = (input[i] > (3+offset1));
int t1_1_0 = rt1_1_0 * (4 + t1_0_1);
t1_1_0 += (1 - rt1_1_0) * t1_0_0;

int rt1_1_1 = (input[i] > (11+offset1));
int t1_1_1 = rt1_1_1 * (4 + t1_0_3);
t1_1_1 += (1 - rt1_1_1) * t1_0_2;

int rt1_1_2 = (input[i] > (19+offset1));
int t1_1_2 = rt1_1_2 * (4 + t1_0_5);
t1_1_2 += (1 - rt1_1_2) * t1_0_4;

int rt1_1_3 = (input[i] > (27+offset1));
int t1_1_3 = rt1_1_3 * (4 + t1_0_7);
t1_1_3 += (1 - rt1_1_3) * t1_0_6;

int rt1_1_4 = (input[i] > (35+offset1));
int t1_1_4 = rt1_1_4 * (4 + t1_0_9);
t1_1_4 += (1 - rt1_1_4) * t1_0_8;

int rt1_1_5 = (input[i] > (43+offset1));
int t1_1_5 = rt1_1_5 * (4 + t1_0_11);
t1_1_5 += (1 - rt1_1_5) * t1_0_10;

int rt1_1_6 = (input[i] > (51+offset1));
int t1_1_6 = rt1_1_6 * (4 + t1_0_13);
t1_1_6 += (1 - rt1_1_6) * t1_0_12;

int rt1_1_7 = (input[i] > (59+offset1));
int t1_1_7 = rt1_1_7 * (4 + t1_0_15);
t1_1_7 += (1 - rt1_1_7) * t1_0_14;

int rt1_1_8 = (input[i] > (67+offset1));
int t1_1_8 = rt1_1_8 * (4 + t1_0_17);
t1_1_8 += (1 - rt1_1_8) * t1_0_16;

int rt1_1_9 = (input[i] > (75+offset1));
int t1_1_9 = rt1_1_9 * (4 + t1_0_19);
t1_1_9 += (1 - rt1_1_9) * t1_0_18;

int rt1_1_10 = (input[i] > (83+offset1));
int t1_1_10 = rt1_1_10 * (4 + t1_0_21);
t1_1_10 += (1 - rt1_1_10) * t1_0_20;

int rt1_1_11 = (input[i] > (91+offset1));
int t1_1_11 = rt1_1_11 * (4 + t1_0_23);
t1_1_11 += (1 - rt1_1_11) * t1_0_22;

int rt1_1_12 = (input[i] > (99+offset1));
int t1_1_12 = rt1_1_12 * (4 + t1_0_25);
t1_1_12 += (1 - rt1_1_12) * t1_0_24;

int rt1_1_13 = (input[i] > (107+offset1));
int t1_1_13 = rt1_1_13 * (4 + t1_0_27);
t1_1_13 += (1 - rt1_1_13) * t1_0_26;

int rt1_1_14 = (input[i] > (115+offset1));
int t1_1_14 = rt1_1_14 * (4 + t1_0_29);
t1_1_14 += (1 - rt1_1_14) * t1_0_28;

int rt1_1_15 = (input[i] > (123+offset1));
int t1_1_15 = rt1_1_15 * (4 + t1_0_31);
t1_1_15 += (1 - rt1_1_15) * t1_0_30;

int rt1_2_0 = (input[i] > (7+offset1));
int t1_2_0 = rt1_2_0 * (8 + t1_1_1);
t1_2_0 += (1 - rt1_2_0) * t1_1_0;

int rt1_2_1 = (input[i] > (23+offset1));
int t1_2_1 = rt1_2_1 * (8 + t1_1_3);
t1_2_1 += (1 - rt1_2_1) * t1_1_2;

int rt1_2_2 = (input[i] > (39+offset1));
int t1_2_2 = rt1_2_2 * (8 + t1_1_5);
t1_2_2 += (1 - rt1_2_2) * t1_1_4;

int rt1_2_3 = (input[i] > (55+offset1));
int t1_2_3 = rt1_2_3 * (8 + t1_1_7);
t1_2_3 += (1 - rt1_2_3) * t1_1_6;

int rt1_2_4 = (input[i] > (71+offset1));
int t1_2_4 = rt1_2_4 * (8 + t1_1_9);
t1_2_4 += (1 - rt1_2_4) * t1_1_8;

int rt1_2_5 = (input[i] > (87+offset1));
int t1_2_5 = rt1_2_5 * (8 + t1_1_11);
t1_2_5 += (1 - rt1_2_5) * t1_1_10;

int rt1_2_6 = (input[i] > (103+offset1));
int t1_2_6 = rt1_2_6 * (8 + t1_1_13);
t1_2_6 += (1 - rt1_2_6) * t1_1_12;

int rt1_2_7 = (input[i] > (119+offset1));
int t1_2_7 = rt1_2_7 * (8 + t1_1_15);
t1_2_7 += (1 - rt1_2_7) * t1_1_14;

int rt1_3_0 = (input[i] > (15+offset1));
int t1_3_0 = rt1_3_0 * (16 + t1_2_1);
t1_3_0 += (1 - rt1_3_0) * t1_2_0;

int rt1_3_1 = (input[i] > (47+offset1));
int t1_3_1 = rt1_3_1 * (16 + t1_2_3);
t1_3_1 += (1 - rt1_3_1) * t1_2_2;

int rt1_3_2 = (input[i] > (79+offset1));
int t1_3_2 = rt1_3_2 * (16 + t1_2_5);
t1_3_2 += (1 - rt1_3_2) * t1_2_4;

int rt1_3_3 = (input[i] > (111+offset1));
int t1_3_3 = rt1_3_3 * (16 + t1_2_7);
t1_3_3 += (1 - rt1_3_3) * t1_2_6;

int rt1_4_0 = (input[i] > (31+offset1));
int t1_4_0 = rt1_4_0 * (32 + t1_3_1);
t1_4_0 += (1 - rt1_4_0) * t1_3_0;

int rt1_4_1 = (input[i] > (95+offset1));
int t1_4_1 = rt1_4_1 * (32 + t1_3_3);
t1_4_1 += (1 - rt1_4_1) * t1_3_2;

int rt1_5_0 = (input[i] > (63+offset1));
int t1_5_0 = rt1_5_0 * (64 + t1_4_1);
t1_5_0 += (1 - rt1_5_0) * t1_4_0;

vR[i] = t0_5_0+ (t0_5_0 == t1_5_0);
	}
}

int main(int argc, char **argv)
{
	float elapsed_time;
	// set up device
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("[CUDA - Random Forest]: EXP %s\n", EXP_NAME);
	printf("[CUDA - Random Forest]: Using Device %d: %s\n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));

	// set up data size of vectors
	int nElem = N_ELEM;
	printf("[CUDA - Random Forest]: Vector Size %d\n", nElem);

	// malloc host memory
	size_t nBytes = nElem * sizeof(float);
	float *h_vR;
	h_vR = (float *)malloc(nBytes);
	float *h_input;
	h_input = (float *)malloc(nBytes);

	printf("[CUDA - Random Forest]: Start Reading Dataset\n");
	initData(h_input, nElem);
	printf("[CUDA - Random Forest]: Finished Reading Dataset\n");
	memset(h_vR, 0, nBytes);

	// malloc device global memory
	float *d_vR;
	CHECK(hipMalloc((float **)&d_vR, nBytes));
	float *d_input;
	CHECK(hipMalloc((float **)&d_input, nBytes));

	// transfer data from host to device
	CHECK(hipMemcpy(d_input, h_input, nBytes, hipMemcpyHostToDevice));

	// invoke kernel at host side
	int iLen = 512;
	dim3 block(iLen);
	dim3 grid((nElem + block.x - 1) / block.x);
	hipEvent_t start, stop;
	CHECK(hipEventCreate(&start));
	CHECK(hipEventCreate(&stop));

	// record start event
	CHECK(hipEventRecord(start, 0));
#if defined(_2TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, d_vR);
#elif defined(_3TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, d_vR);
#elif defined(_4TREE)
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, d_vR);
#else
	RF_kernel<<<grid, block>>>(d_input, nElem, 0, d_vR);
#endif
	CHECK(hipEventRecord(stop, 0));
	CHECK(hipEventSynchronize(stop));
	hipDeviceSynchronize();

	// calculate elapsed time
	CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
	printf("[CUDA - Random Forest]: %s - execution time = %.6fms\n", EXP_NAME, elapsed_time);
	registerTime(elapsed_time);

	// copy kernel result back to host side
	CHECK(hipMemcpy(h_vR, d_vR, nBytes, hipMemcpyDeviceToHost));
	CHECK(hipGetLastError());

	//Verify the answers
	char flag = 1;
	for (int i = 0; i < nElem; i++)
	{
		if (i < 32)
		{
			printf("%.3f ", h_vR[i]);
		}
		if (h_input[i] != h_vR[i])
		{
			flag = 0;
			break;
		}
	}
	if (flag == 1)
	{
//		printf("Return correct");
	}
	else
	{
//		printf("Error!!");
	}
	printf("\n ");

	// free device memory
	hipFree(d_input);
	hipFree(d_vR);

	// free host memory
	free(h_input);
	free(h_vR);
	return (0);
}

void initData(float *ip, int size)
{
	for (int i = 0; i < size; i++)
	{
		ip[i] = (float)(0);
	}
	return;
}

void registerTime(float value)
{
	printf("[CUDA - Random Forest]: Time spent: %f.\n", value);
}
