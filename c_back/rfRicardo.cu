#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>

#define WARP_SIZE 32
#define N_ELEM 200000000

#define CHECK(call)                                                \
    {                                                              \
        const hipError_t error = call;                            \
        if (error != hipSuccess)                                  \
        {                                                          \
            fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
            fprintf(stderr, "code: %d, reason: %s\n", error,       \
                    hipGetErrorString(error));                    \
        }                                                          \
    }
FILE *outFile;

// Type of experiment
// #define _RF_WITH_CALC_1TREE_1INPUT_3LEVEL
// #define _RF_WITH_CALC_1TREE_1INPUT_5LEVEL
#define _RF_WITH_CALC_1TREE_1INPUT_7LEVEL
// #define _RF_WITH_IF_1TREE_1INPUT_3LEVEL_BEST
// #define _RF_WITH_IF_1TREE_1INPUT_3LEVEL_WORST
// #define _RF_WITH_IF_1TREE_1INPUT_7LEVEL_BEST
// #define _RF_WITH_IF_1TREE_1INPUT_7LEVEL_WORST
// #define _RF_WITH_IF_4TREE_1INPUT_5LEVEL_BEST
// #define _RF_WITH_IF_4TREE_1INPUT_5LEVEL_WORST


// IF
#if defined(_RF_WITH_IF_1TREE_1INPUT_3LEVEL_BEST) || defined(_RF_WITH_IF_1TREE_1INPUT_3LEVEL_WORST)

#define _1_INPUT

#ifdef _RF_WITH_IF_1TREE_1INPUT_3LEVEL_BEST
#define EXP_NAME "_RF_WITH_IF_1TREE_1INPUT_3LEVEL_BEST"
#define EXP_OUTPUT_FILE "_RF_WITH_IF_1TREE_1INPUT_3LEVEL_BEST_OUTPUT.txt"
#define EXP_RESULT_FILE "_RF_WITH_IF_1TREE_1INPUT_3LEVEL_BEST_RESULT.csv"
#endif

#ifdef _RF_WITH_IF_1TREE_1INPUT_3LEVEL_WORST
#define EXP_NAME "_RF_WITH_IF_1TREE_1INPUT_3LEVEL_WORST"
#define EXP_OUTPUT_FILE "_RF_WITH_IF_1TREE_1INPUT_3LEVEL_WORST_OUTPUT.txt"
#define EXP_RESULT_FILE "_RF_WITH_IF_1TREE_1INPUT_3LEVEL_WORST_RESULT.csv"
#endif

// RF_with_calc
__global__ void RF_kernel(float *input, const int n, float *vR)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // para 8 classes
    if (i < n)
    {
        if (input[i] > 3)
        {
            if (input[i] > 5)
            {
                if (input[i] == 7)
                {
                    vR[i] = 7;
                }
                else
                { //!=7
                    vR[i] = 6;
                }
            }
            else
            { // <=5
                if (input[i] == 5)
                {
                    vR[i] = 5;
                }
                else
                { //!=5
                    vR[i] = 4;
                }
            }
        }
        else
        { //<=3
            if (input[i] > 1)
            {
                if (input[i] == 3)
                {
                    vR[i] = 3;
                }
                else
                { //!=3
                    vR[i] = 2;
                }
            }
            else
            { // <=1
                if (input[i] == 1)
                {
                    vR[i] = 1;
                }
                else
                { //!=1
                    vR[i] = 0;
                }
            }
        }
    }
}

#endif

#if defined(_RF_WITH_IF_1TREE_1INPUT_7LEVEL_BEST) || defined(_RF_WITH_IF_1TREE_1INPUT_7LEVEL_WORST)

#define _1_INPUT

#ifdef _RF_WITH_IF_1TREE_1INPUT_7LEVEL_BEST
#define EXP_NAME "_RF_WITH_IF_1TREE_1INPUT_7LEVEL_BEST"
#define EXP_OUTPUT_FILE "_RF_WITH_IF_1TREE_1INPUT_7LEVEL_BEST_OUTPUT.txt"
#define EXP_RESULT_FILE "_RF_WITH_IF_1TREE_1INPUT_7LEVEL_BEST_RESULT.csv"
#endif

#ifdef _RF_WITH_IF_1TREE_1INPUT_7LEVEL_WORST
#define EXP_NAME "_RF_WITH_IF_1TREE_1INPUT_7LEVEL_WORST"
#define EXP_OUTPUT_FILE "_RF_WITH_IF_1TREE_1INPUT_7LEVEL_WORST_OUTPUT.txt"
#define EXP_RESULT_FILE "_RF_WITH_IF_1TREE_1INPUT_7LEVEL_WORST_RESULT.csv"

#endif

// RF_with_calc
__global__ void RF_kernel(float *input, const int n, float *vR)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // para 128 classes
    if (i < n)
    {
        if (input[i] > 63)
        {
            if (input[i] > 95)
            {
                if (input[i] > 111)
                {
                    if (input[i] > 119)
                    {
                        if (input[i] > 123)
                        {
                            if (input[i] > 125)
                            {
                                if (input[i] == 127)
                                {
                                    vR[i] = 127;
                                }
                                else
                                { //!=127
                                    vR[i] = 126;
                                }
                            }
                            else
                            { // <= 125
                                if (input[i] == 125)
                                {
                                    vR[i] = 125;
                                }
                                else
                                { //!=125
                                    vR[i] = 124;
                                }
                            }
                        }
                        else
                        { //<=123
                            if (input[i] > 121)
                            {
                                if (input[i] == 123)
                                {
                                    vR[i] = 123;
                                }
                                else
                                { //!=123
                                    vR[i] = 122;
                                }
                            }
                            else
                            { // <=121
                                if (input[i] == 12)
                                {
                                    vR[i] = 121;
                                }
                                else
                                { //!=121
                                    vR[i] = 120;
                                }
                            }
                        }
                    }
                    else
                    { //<=119
                        if (input[i] > 115)
                        {
                            if (input[i] > 117)
                            {
                                if (input[i] == 119)
                                {
                                    vR[i] = 119;
                                }
                                else
                                { //!=119
                                    vR[i] = 118;
                                }
                            }
                            else
                            { // <=117
                                if (input[i] == 17)
                                {
                                    vR[i] = 117;
                                }
                                else
                                { //!=117
                                    vR[i] = 116;
                                }
                            }
                        }
                        else
                        { //<=115
                            if (input[i] > 113)
                            {
                                if (input[i] == 115)
                                {
                                    vR[i] = 115;
                                }
                                else
                                { //!=115
                                    vR[i] = 114;
                                }
                            }
                            else
                            { // <=113
                                if (input[i] == 113)
                                {
                                    vR[i] = 113;
                                }
                                else
                                { //!=113
                                    vR[i] = 112;
                                }
                            }
                        }
                    }
                }
                else
                { //<=111
                    if (input[i] > 103)
                    {
                        if (input[i] > 107)
                        {
                            if (input[i] > 109)
                            {
                                if (input[i] == 111)
                                {
                                    vR[i] = 111;
                                }
                                else
                                { //!=111
                                    vR[i] = 110;
                                }
                            }
                            else
                            { // <=109
                                if (input[i] == 109)
                                {
                                    vR[i] = 109;
                                }
                                else
                                { //!=109
                                    vR[i] = 108;
                                }
                            }
                        }
                        else
                        {
                            if (input[i] > 105)
                            {
                                if (input[i] == 107)
                                {
                                    vR[i] = 107;
                                }
                                else
                                { //!=107
                                    vR[i] = 106;
                                }
                            }
                            else
                            { //<=105
                                if (input[i] == 105)
                                {
                                    vR[i] = 105;
                                }
                                else
                                { //!=105
                                    vR[i] = 104;
                                }
                            }
                        }
                    }
                    else
                    {
                        if (input[i] > 99)
                        {
                            if (input[i] > 101)
                            {
                                if (input[i] == 103)
                                {
                                    vR[i] = 103;
                                }
                                else
                                { //!=103
                                    vR[i] = 102;
                                }
                            }
                            else
                            { // <=101
                                if (input[i] == 101)
                                {
                                    vR[i] = 101;
                                }
                                else
                                { //!=101
                                    vR[i] = 100;
                                }
                            }
                        }
                        else
                        {
                            if (input[i] > 97)
                            {
                                if (input[i] == 99)
                                {
                                    vR[i] = 99;
                                }
                                else
                                { //!=99
                                    vR[i] = 98;
                                }
                            }
                            else
                            { // <=97
                                if (input[i] == 97)
                                {
                                    vR[i] = 97;
                                }
                                else
                                { //!=97
                                    vR[i] = 96;
                                }
                            }
                        }
                    }
                }
            }
            else
            { //<=95
                if (input[i] > 79)
                {
                    if (input[i] > 87)
                    {
                        if (input[i] > 91)
                        {
                            if (input[i] > 93)
                            {
                                if (input[i] == 95)
                                {
                                    vR[i] = 95;
                                }
                                else
                                { //!=95
                                    vR[i] = 94;
                                }
                            }
                            else
                            { //<=93
                                if (input[i] == 93)
                                {
                                    vR[i] = 93;
                                }
                                else
                                { //!=93
                                    vR[i] = 92;
                                }
                            }
                        }
                        else
                        { //<=91
                            if (input[i] > 89)
                            {
                                if (input[i] == 91)
                                {
                                    vR[i] = 91;
                                }
                                else
                                { //!=91
                                    vR[i] = 90;
                                }
                            }
                            else
                            { // <=89
                                if (input[i] == 89)
                                {
                                    vR[i] = 89;
                                }
                                else
                                { //!=89
                                    vR[i] = 88;
                                }
                            }
                        }
                    }
                    else
                    { //<=87
                        if (input[i] > 83)
                        {
                            if (input[i] > 85)
                            {
                                if (input[i] == 87)
                                {
                                    vR[i] = 87;
                                }
                                else
                                { //!=87
                                    vR[i] = 86;
                                }
                            }
                            else
                            { // <=85
                                if (input[i] == 85)
                                {
                                    vR[i] = 85;
                                }
                                else
                                { //!=85
                                    vR[i] = 84;
                                }
                            }
                        }
                        else
                        { //<=83
                            if (input[i] > 81)
                            {
                                if (input[i] == 83)
                                {
                                    vR[i] = 83;
                                }
                                else
                                { //!=83
                                    vR[i] = 82;
                                }
                            }
                            else
                            { // <=81
                                if (input[i] == 81)
                                {
                                    vR[i] = 81;
                                }
                                else
                                { //!=81
                                    vR[i] = 80;
                                }
                            }
                        }
                    }
                }
                else
                { //<=79
                    if (input[i] > 71)
                    {
                        if (input[i] > 75)
                        {
                            if (input[i] > 77)
                            {
                                if (input[i] == 79)
                                {
                                    vR[i] = 79;
                                }
                                else
                                { //!=79
                                    vR[i] = 78;
                                }
                            }
                            else
                            { // <= 77
                                if (input[i] == 77)
                                {
                                    vR[i] = 77;
                                }
                                else
                                { //!=77
                                    vR[i] = 76;
                                }
                            }
                        }
                        else
                        { //<=75
                            if (input[i] > 73)
                            {
                                if (input[i] == 75)
                                {
                                    vR[i] = 75;
                                }
                                else
                                { //!=75
                                    vR[i] = 74;
                                }
                            }
                            else
                            { // <=73
                                if (input[i] == 73)
                                {
                                    vR[i] = 73;
                                }
                                else
                                { //!=73
                                    vR[i] = 72;
                                }
                            }
                        }
                    }
                    else
                    { // <=71
                        if (input[i] > 67)
                        {
                            if (input[i] > 69)
                            {
                                if (input[i] == 71)
                                {
                                    vR[i] = 71;
                                }
                                else
                                { //!=71
                                    vR[i] = 70;
                                }
                            }
                            else
                            { // <=69
                                if (input[i] == 69)
                                {
                                    vR[i] = 69;
                                }
                                else
                                { //!=69
                                    vR[i] = 68;
                                }
                            }
                        }
                        else
                        { // <=67
                            if (input[i] > 65)
                            {
                                if (input[i] == 67)
                                {
                                    vR[i] = 67;
                                }
                                else
                                { //!=67
                                    vR[i] = 66;
                                }
                            }
                            else
                            { // <=65
                                if (input[i] == 65)
                                {
                                    vR[i] = 65;
                                }
                                else
                                { //!=65
                                    vR[i] = 64;
                                }
                            }
                        }
                    }
                }
            }
        }
        else
        { //<=63
            if (input[i] > 31)
            {
                if (input[i] > 47)
                {
                    if (input[i] > 55)
                    {
                        if (input[i] > 59)
                        {
                            if (input[i] > 61)
                            {
                                if (input[i] == 63)
                                {
                                    vR[i] = 63;
                                }
                                else
                                { //!=63
                                    vR[i] = 62;
                                }
                            }
                            else
                            { // <=61
                                if (input[i] == 61)
                                {
                                    vR[i] = 61;
                                }
                                else
                                { //!=61
                                    vR[i] = 60;
                                }
                            }
                        }
                        else
                        { //<=59
                            if (input[i] > 57)
                            {
                                if (input[i] == 59)
                                {
                                    vR[i] = 59;
                                }
                                else
                                { //!=59
                                    vR[i] = 58;
                                }
                            }
                            else
                            { // <=57
                                if (input[i] == 57)
                                {
                                    vR[i] = 57;
                                }
                                else
                                { //!=57
                                    vR[i] = 56;
                                }
                            }
                        }
                    }
                    else
                    { //<=55
                        if (input[i] > 51)
                        {
                            if (input[i] > 53)
                            {
                                if (input[i] == 55)
                                {
                                    vR[i] = 55;
                                }
                                else
                                { //!=55
                                    vR[i] = 54;
                                }
                            }
                            else
                            { // <=53
                                if (input[i] == 53)
                                {
                                    vR[i] = 53;
                                }
                                else
                                { //!=53
                                    vR[i] = 52;
                                }
                            }
                        }
                        else
                        { //<=51
                            if (input[i] > 49)
                            {
                                if (input[i] == 51)
                                {
                                    vR[i] = 51;
                                }
                                else
                                { //!=51
                                    vR[i] = 50;
                                }
                            }
                            else
                            { // <=49
                                if (input[i] == 49)
                                {
                                    vR[i] = 49;
                                }
                                else
                                { //!=49
                                    vR[i] = 48;
                                }
                            }
                        }
                    }
                }
                else
                { //<=47
                    if (input[i] > 39)
                    {
                        if (input[i] > 43)
                        {
                            if (input[i] > 45)
                            {
                                if (input[i] == 47)
                                {
                                    vR[i] = 47;
                                }
                                else
                                { //!=47
                                    vR[i] = 46;
                                }
                            }
                            else
                            { // <=45
                                if (input[i] == 45)
                                {
                                    vR[i] = 45;
                                }
                                else
                                { //!=45
                                    vR[i] = 44;
                                }
                            }
                        }
                        else
                        { //<=43
                            if (input[i] > 41)
                            {
                                if (input[i] == 43)
                                {
                                    vR[i] = 43;
                                }
                                else
                                { //!=43
                                    vR[i] = 42;
                                }
                            }
                            else
                            { // <=41
                                if (input[i] == 41)
                                {
                                    vR[i] = 41;
                                }
                                else
                                { //!=41
                                    vR[i] = 40;
                                }
                            }
                        }
                    }
                    else
                    { //<=39
                        if (input[i] > 35)
                        {
                            if (input[i] > 37)
                            {
                                if (input[i] == 39)
                                {
                                    vR[i] = 39;
                                }
                                else
                                { //!=39
                                    vR[i] = 38;
                                }
                            }
                            else
                            { // <=37
                                if (input[i] == 37)
                                {
                                    vR[i] = 37;
                                }
                                else
                                { //!=37
                                    vR[i] = 36;
                                }
                            }
                        }
                        else
                        { //<=35
                            if (input[i] > 33)
                            {
                                if (input[i] == 35)
                                {
                                    vR[i] = 35;
                                }
                                else
                                { //!=35
                                    vR[i] = 34;
                                }
                            }
                            else
                            { // <=33
                                if (input[i] == 33)
                                {
                                    vR[i] = 33;
                                }
                                else
                                { //!=35
                                    vR[i] = 32;
                                }
                            }
                        }
                    }
                }
            }
            else
            { //<=31
                if (input[i] > 15)
                {
                    if (input[i] > 23)
                    {
                        if (input[i] > 27)
                        {
                            if (input[i] > 29)
                            {
                                if (input[i] == 31)
                                {
                                    vR[i] = 31;
                                }
                                else
                                { //!=31
                                    vR[i] = 30;
                                }
                            }
                            else
                            { // <=29
                                if (input[i] == 29)
                                {
                                    vR[i] = 29;
                                }
                                else
                                { //!=29
                                    vR[i] = 28;
                                }
                            }
                        }
                        else
                        { //<=27
                            if (input[i] > 25)
                            {
                                if (input[i] == 27)
                                {
                                    vR[i] = 27;
                                }
                                else
                                { //!=27
                                    vR[i] = 26;
                                }
                            }
                            else
                            { // <=25
                                if (input[i] == 25)
                                {
                                    vR[i] = 25;
                                }
                                else
                                { //!=25
                                    vR[i] = 24;
                                }
                            }
                        }
                    }
                    else
                    { //<=23
                        if (input[i] > 19)
                        {
                            if (input[i] > 21)
                            {
                                if (input[i] == 23)
                                {
                                    vR[i] = 23;
                                }
                                else
                                { //!=23
                                    vR[i] = 22;
                                }
                            }
                            else
                            { // <=21
                                if (input[i] == 21)
                                {
                                    vR[i] = 21;
                                }
                                else
                                { //!=21
                                    vR[i] = 20;
                                }
                            }
                        }
                        else
                        { //<=19
                            if (input[i] > 17)
                            {
                                if (input[i] == 19)
                                {
                                    vR[i] = 19;
                                }
                                else
                                { //!=19
                                    vR[i] = 18;
                                }
                            }
                            else
                            { //<=17
                                if (input[i] == 17)
                                {
                                    vR[i] = 17;
                                }
                                else
                                { //!=
                                    vR[i] = 16;
                                }
                            }
                        }
                    }
                }
                else
                { //<=15
                    if (input[i] > 7)
                    {
                        if (input[i] > 11)
                        {
                            if (input[i] > 13)
                            {
                                if (input[i] == 15)
                                {
                                    vR[i] = 15;
                                }
                                else
                                { //!=15
                                    vR[i] = 14;
                                }
                            }
                            else
                            { // <=13
                                if (input[i] == 13)
                                {
                                    vR[i] = 13;
                                }
                                else
                                { //!=13
                                    vR[i] = 12;
                                }
                            }
                        }
                        else
                        { //<=11
                            if (input[i] > 9)
                            {
                                if (input[i] == 11)
                                {
                                    vR[i] = 11;
                                }
                                else
                                { //!=11
                                    vR[i] = 10;
                                }
                            }
                            else
                            { //<=9
                                if (input[i] == 9)
                                {
                                    vR[i] = 9;
                                }
                                else
                                { //!=9
                                    vR[i] = 8;
                                }
                            }
                        }
                    }
                    else
                    { //<=7
                        if (input[i] > 3)
                        {
                            if (input[i] > 5)
                            {
                                if (input[i] == 7)
                                {
                                    vR[i] = 7;
                                }
                                else
                                { //!=7
                                    vR[i] = 6;
                                }
                            }
                            else
                            { // <=5
                                if (input[i] == 5)
                                {
                                    vR[i] = 5;
                                }
                                else
                                { //!=5
                                    vR[i] = 4;
                                }
                            }
                        }
                        else
                        { //<=3
                            if (input[i] > 1)
                            {
                                if (input[i] == 3)
                                {
                                    vR[i] = 3;
                                }
                                else
                                { //!=3
                                    vR[i] = 2;
                                }
                            }
                            else
                            { // <=1
                                if (input[i] == 1)
                                {
                                    vR[i] = 1;
                                }
                                else
                                { //!=1
                                    vR[i] = 0;
                                }
                            }
                        }
                    }
                }
            }
        }
    }
}

#endif

#if defined(_RF_WITH_IF_4TREE_1INPUT_5LEVEL_BEST) || defined(_RF_WITH_IF_4TREE_1INPUT_5LEVEL_WORST)

#define _1_INPUT

#ifdef _RF_WITH_IF_4TREE_1INPUT_5LEVEL_BEST
#define EXP_NAME "_RF_WITH_IF_4TREE_1INPUT_5LEVEL_BEST"
#define EXP_OUTPUT_FILE "_RF_WITH_IF_4TREE_1INPUT_5LEVEL_BEST_OUTPUT.txt"
#define EXP_RESULT_FILE "_RF_WITH_IF_4TREE_1INPUT_5LEVEL_BEST_RESULT.csv"
#endif

#ifdef _RF_WITH_IF_4TREE_1INPUT_5LEVEL_WORST
#define EXP_NAME "_RF_WITH_IF_4TREE_1INPUT_5LEVEL_WORST"
#define EXP_OUTPUT_FILE "_RF_WITH_IF_4TREE_1INPUT_5LEVEL_WORST_OUTPUT.txt"
#define EXP_RESULT_FILE "_RF_WITH_IF_4TREE_1INPUT_5LEVEL_WORST_RESULT.csv"

#endif

// RF_with_calc
__global__ void RF_kernel(float *input, const int n, float *vR)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // para 32 classes
    if (i < n)
    {
        if (input[i] > 15)
        {
            if (input[i] > 23)
            {
                if (input[i] > 27)
                {
                    if (input[i] > 29)
                    {
                        if (input[i] == 31)
                        {
                            vR[i] = 31;
                        }
                        else
                        { //!=31
                            vR[i] = 30;
                        }
                    }
                    else
                    { //<=29
                        if (input[i] == 29)
                        {
                            vR[i] = 29;
                        }
                        else
                        { //!=29
                            vR[i] = 28;
                        }
                    }
                }
                else
                { //<=27
                    if (input[i] > 25)
                    {
                        if (input[i] == 27)
                        {
                            vR[i] = 27;
                        }
                        else
                        { //!=27
                            vR[i] = 26;
                        }
                    }
                    else
                    { //<=25
                        if (input[i] == 25)
                        {
                            vR[i] = 25;
                        }
                        else
                        { //!=25
                            vR[i] = 24;
                        }
                    }
                }
            }
            else
            { //<=23
                if (input[i] > 19)
                {
                    if (input[i] > 21)
                    {
                        if (input[i] == 23)
                        {
                            vR[i] = 23;
                        }
                        else
                        { //!=23
                            vR[i] = 22;
                        }
                    }
                    else
                    { //<=21
                        if (input[i] == 21)
                        {
                            vR[i] = 21;
                        }
                        else
                        { //!=21
                            vR[i] = 20;
                        }
                    }
                }
                else
                { //<=19
                    if (input[i] > 17)
                    {
                        if (input[i] == 19)
                        {
                            vR[i] = 19;
                        }
                        else
                        { // !=19
                            vR[i] = 18;
                        }
                    }
                    else
                    { //<=17
                        if (input[i] == 17)
                        {
                            vR[i] = 17;
                        }
                        else
                        { //!=17
                            vR[i] = 16;
                        }
                    }
                }
            }
        }
        else
        { // <=15
            if (input[i] > 7)
            {
                if (input[i] > 11)
                {
                    if (input[i] > 13)
                    {
                        if (input[i] == 15)
                        {
                            vR[i] = 15;
                        }
                        else
                        { //!=15
                            vR[i] = 14;
                        }
                    }
                    else
                    { //<=13
                        if (input[i] == 13)
                        {
                            vR[i] = 13;
                        }
                        else
                        { //!=13
                            vR[i] = 12;
                        }
                    }
                }
                else
                { //<=11
                    if (input[i] > 9)
                    {
                        if (input[i] == 11)
                        {
                            vR[i] = 11;
                        }
                        else
                        { //!=11
                            vR[i] = 10;
                        }
                    }
                    else
                    { //<=9
                        if (input[i] == 9)
                        {
                            vR[i] = 9;
                        }
                        else
                        { //!=9
                            vR[i] = 8;
                        }
                    }
                }
            }
            else
            { //<=7
                if (input[i] > 3)
                {
                    if (input[i] > 5)
                    {
                        if (input[i] == 7)
                        {
                            vR[i] = 7;
                        }
                        else
                        { //!=7
                            vR[i] = 6;
                        }
                    }
                    else
                    { //<=5
                        if (input[i] == 5)
                        {
                            vR[i] = 5;
                        }
                        else
                        { //!=5
                            vR[i] = 4;
                        }
                    }
                }
                else
                { //<=3
                    if (input[i] > 1)
                    {
                        if (input[i] == 3)
                        {
                            vR[i] = 3;
                        }
                        else
                        { // !=3
                            vR[i] = 2;
                        }
                    }
                    else
                    { //<=1
                        if (input[i] == 1)
                        {
                            vR[i] = 1;
                        }
                        else
                        { //!=1
                            vR[i] = 0;
                        }
                    }
                }
            }
        }

        if (input[i] > 15)
        {
            if (input[i] > 23)
            {
                if (input[i] > 27)
                {
                    if (input[i] > 29)
                    {
                        if (input[i] == 31)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=31
                            vR[i] += 1;
                        }
                    }
                    else
                    { //<=29
                        if (input[i] == 29)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=29
                            vR[i] += 1;
                        }
                    }
                }
                else
                { //<=27
                    if (input[i] > 25)
                    {
                        if (input[i] == 27)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=27
                            vR[i] += 1;
                        }
                    }
                    else
                    { //<=25
                        if (input[i] == 25)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=25
                            vR[i] += 1;
                        }
                    }
                }
            }
            else
            { //<=23
                if (input[i] > 19)
                {
                    if (input[i] > 21)
                    {
                        if (input[i] == 23)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=23
                            vR[i] += 1;
                        }
                    }
                    else
                    { //<=21
                        if (input[i] == 21)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=21
                            vR[i] += 1;
                        }
                    }
                }
                else
                { //<=19
                    if (input[i] > 17)
                    {
                        if (input[i] == 19)
                        {
                            vR[i] += 1;
                        }
                        else
                        { // !=19
                            vR[i] += 1;
                        }
                    }
                    else
                    { //<=17
                        if (input[i] == 17)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=17
                            vR[i] += 1;
                        }
                    }
                }
            }
        }
        else
        { // <=15
            if (input[i] > 7)
            {
                if (input[i] > 11)
                {
                    if (input[i] > 13)
                    {
                        if (input[i] == 15)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=15
                            vR[i] += 1;
                        }
                    }
                    else
                    { //<=13
                        if (input[i] == 13)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=13
                            vR[i] += 1;
                        }
                    }
                }
                else
                { //<=11
                    if (input[i] > 9)
                    {
                        if (input[i] == 11)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=11
                            vR[i] += 1;
                        }
                    }
                    else
                    { //<=9
                        if (input[i] == 9)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=9
                            vR[i] += 1;
                        }
                    }
                }
            }
            else
            { //<=7
                if (input[i] > 3)
                {
                    if (input[i] > 5)
                    {
                        if (input[i] == 7)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=7
                            vR[i] += 1;
                        }
                    }
                    else
                    { //<=5
                        if (input[i] == 5)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=5
                            vR[i] += 1;
                        }
                    }
                }
                else
                { //<=3
                    if (input[i] > 1)
                    {
                        if (input[i] == 3)
                        {
                            vR[i] += 1;
                        }
                        else
                        { // !=3
                            vR[i] += 1;
                        }
                    }
                    else
                    { //<=1
                        if (input[i] == 1)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=1
                            vR[i] += 1;
                        }
                    }
                }
            }
        }

        if (input[i] > 15)
        {
            if (input[i] > 23)
            {
                if (input[i] > 27)
                {
                    if (input[i] > 29)
                    {
                        if (input[i] == 31)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=31
                            vR[i] += 1;
                        }
                    }
                    else
                    { //<=29
                        if (input[i] == 29)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=29
                            vR[i] += 1;
                        }
                    }
                }
                else
                { //<=27
                    if (input[i] > 25)
                    {
                        if (input[i] == 27)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=27
                            vR[i] += 1;
                        }
                    }
                    else
                    { //<=25
                        if (input[i] == 25)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=25
                            vR[i] += 1;
                        }
                    }
                }
            }
            else
            { //<=23
                if (input[i] > 19)
                {
                    if (input[i] > 21)
                    {
                        if (input[i] == 23)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=23
                            vR[i] += 1;
                        }
                    }
                    else
                    { //<=21
                        if (input[i] == 21)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=21
                            vR[i] += 1;
                        }
                    }
                }
                else
                { //<=19
                    if (input[i] > 17)
                    {
                        if (input[i] == 19)
                        {
                            vR[i] += 1;
                        }
                        else
                        { // !=19
                            vR[i] += 1;
                        }
                    }
                    else
                    { //<=17
                        if (input[i] == 17)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=17
                            vR[i] += 1;
                        }
                    }
                }
            }
        }
        else
        { // <=15
            if (input[i] > 7)
            {
                if (input[i] > 11)
                {
                    if (input[i] > 13)
                    {
                        if (input[i] == 15)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=15
                            vR[i] += 1;
                        }
                    }
                    else
                    { //<=13
                        if (input[i] == 13)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=13
                            vR[i] += 1;
                        }
                    }
                }
                else
                { //<=11
                    if (input[i] > 9)
                    {
                        if (input[i] == 11)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=11
                            vR[i] += 1;
                        }
                    }
                    else
                    { //<=9
                        if (input[i] == 9)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=9
                            vR[i] += 1;
                        }
                    }
                }
            }
            else
            { //<=7
                if (input[i] > 3)
                {
                    if (input[i] > 5)
                    {
                        if (input[i] == 7)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=7
                            vR[i] += 1;
                        }
                    }
                    else
                    { //<=5
                        if (input[i] == 5)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=5
                            vR[i] += 1;
                        }
                    }
                }
                else
                { //<=3
                    if (input[i] > 1)
                    {
                        if (input[i] == 3)
                        {
                            vR[i] += 1;
                        }
                        else
                        { // !=3
                            vR[i] += 1;
                        }
                    }
                    else
                    { //<=1
                        if (input[i] == 1)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=1
                            vR[i] += 1;
                        }
                    }
                }
            }
        }

        if (input[i] > 15)
        {
            if (input[i] > 23)
            {
                if (input[i] > 27)
                {
                    if (input[i] > 29)
                    {
                        if (input[i] == 31)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=31
                            vR[i] += 1;
                        }
                    }
                    else
                    { //<=29
                        if (input[i] == 29)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=29
                            vR[i] += 1;
                        }
                    }
                }
                else
                { //<=27
                    if (input[i] > 25)
                    {
                        if (input[i] == 27)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=27
                            vR[i] += 1;
                        }
                    }
                    else
                    { //<=25
                        if (input[i] == 25)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=25
                            vR[i] += 1;
                        }
                    }
                }
            }
            else
            { //<=23
                if (input[i] > 19)
                {
                    if (input[i] > 21)
                    {
                        if (input[i] == 23)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=23
                            vR[i] += 1;
                        }
                    }
                    else
                    { //<=21
                        if (input[i] == 21)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=21
                            vR[i] += 1;
                        }
                    }
                }
                else
                { //<=19
                    if (input[i] > 17)
                    {
                        if (input[i] == 19)
                        {
                            vR[i] += 1;
                        }
                        else
                        { // !=19
                            vR[i] += 1;
                        }
                    }
                    else
                    { //<=17
                        if (input[i] == 17)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=17
                            vR[i] += 1;
                        }
                    }
                }
            }
        }
        else
        { // <=15
            if (input[i] > 7)
            {
                if (input[i] > 11)
                {
                    if (input[i] > 13)
                    {
                        if (input[i] == 15)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=15
                            vR[i] += 1;
                        }
                    }
                    else
                    { //<=13
                        if (input[i] == 13)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=13
                            vR[i] += 1;
                        }
                    }
                }
                else
                { //<=11
                    if (input[i] > 9)
                    {
                        if (input[i] == 11)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=11
                            vR[i] += 1;
                        }
                    }
                    else
                    { //<=9
                        if (input[i] == 9)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=9
                            vR[i] += 1;
                        }
                    }
                }
            }
            else
            { //<=7
                if (input[i] > 3)
                {
                    if (input[i] > 5)
                    {
                        if (input[i] == 7)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=7
                            vR[i] += 1;
                        }
                    }
                    else
                    { //<=5
                        if (input[i] == 5)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=5
                            vR[i] += 1;
                        }
                    }
                }
                else
                { //<=3
                    if (input[i] > 1)
                    {
                        if (input[i] == 3)
                        {
                            vR[i] += 1;
                        }
                        else
                        { // !=3
                            vR[i] += 1;
                        }
                    }
                    else
                    { //<=1
                        if (input[i] == 1)
                        {
                            vR[i] += 1;
                        }
                        else
                        { //!=1
                            vR[i] += 1;
                        }
                    }
                }
            }
        }
    }
}

#endif
//*********

// CALC
#if defined(_RF_WITH_CALC_1TREE_1INPUT_3LEVEL)

#define EXP_NAME "_RF_WITH_CALC_1TREE_1INPUT_3LEVEL"
#define EXP_OUTPUT_FILE "_RF_WITH_CALC_1TREE_1INPUT_3LEVEL.txt"
#define EXP_RESULT_FILE "_RF_WITH_CALC_1TREE_1INPUT_3LEVEL.csv"
#define _1_INPUT

// RF_with_calc
__global__ void RF_kernel(float *input, const int n, float *vR)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // para 8 classes
    if (id < n)
    {
        char rt00 = (input[id] > 1); // root t00
        char t00 = rt00 * (2 + (input[id] > 2));
        t00 += (1 - rt00) * (input[id] > 0); // 0, 1, 2, 3

        char rt01 = (input[id] > 5); // root t01
        char t01 = rt01 * (2 + (input[id] > 6));
        t01 += (1 - rt01) * (input[id] > 4); // 0, 1, 2, 3

        char root = (input[id] > 3); // root 10
        char leaf = root * (4 + t01);
        leaf += (1 - root) * t00; // 0, 1, 2, 3, 4, 5, 6, 7

        vR[id] = leaf;
    }
}

#endif

#if defined(_RF_WITH_CALC_1TREE_1INPUT_5LEVEL)

#define EXP_NAME "_RF_WITH_CALC_1TREE_1INPUT_5LEVEL"
#define EXP_OUTPUT_FILE "_RF_WITH_CALC_1TREE_1INPUT_5LEVEL.txt"
#define EXP_RESULT_FILE "_RF_WITH_CALC_1TREE_1INPUT_5LEVEL.csv"
#define _1_INPUT

// RF_with_calc
__global__ void RF_kernel(float *input, const int n, float *vR)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // para 32 classes
    if (id < n)
    {
        char rt00 = (input[id] > 1); // root t00
        char t00 = rt00 * (2 + (input[id] > 2));
        t00 += (1 - rt00) * (input[id] > 0); // 0, 1, 2, 3

        char rt01 = (input[id] > 5); // root t01
        char t01 = rt01 * (2 + (input[id] > 6));
        t01 += (1 - rt01) * (input[id] > 4); // 0, 1, 2, 3

        char rt02 = (input[id] > 9); // root t02
        char t02 = rt02 * (2 + (input[id] > 10));
        t02 += (1 - rt02) * (input[id] > 8); // 0, 1, 2, 3

        char rt03 = (input[id] > 13); // root t03
        char t03 = rt03 * (2 + (input[id] > 14));
        t03 += (1 - rt03) * (input[id] > 12); // 0, 1, 2, 3

        char rt04 = (input[id] > 17); // root t04
        char t04 = rt04 * (2 + (input[id] > 18));
        t04 += (1 - rt04) * (input[id] > 16); // 0, 1, 2, 3

        char rt05 = (input[id] > 21); // root t05
        char t05 = rt05 * (2 + (input[id] > 22));
        t05 += (1 - rt05) * (input[id] > 20); // 0, 1, 2, 3

        char rt06 = (input[id] > 25); // root t06
        char t06 = rt06 * (2 + (input[id] > 26));
        t06 += (1 - rt06) * (input[id] > 24); // 0, 1, 2, 3

        char rt07 = (input[id] > 29); // root t07
        char t07 = rt07 * (2 + (input[id] > 30));
        t07 += (1 - rt07) * (input[id] > 28); // 0, 1, 2, 3

        char rt10 = (input[id] > 3); // root 10
        char t10 = rt10 * (4 + t01);
        t10 += (1 - rt10) * t00; // 0, 1, 2, 3, 4, 5, 6, 7

        char rt11 = (input[id] > 11); // root 11
        char t11 = rt11 * (4 + t03);
        t11 += (1 - rt11) * t02; // 0, 1, 2, 3, 4, 5, 6, 7

        char rt12 = (input[id] > 19); // root 12
        char t12 = rt12 * (4 + t05);
        t12 += (1 - rt12) * t04; // 0, 1, 2, 3, 4, 5, 6, 7

        char rt13 = (input[id] > 27); // root 13
        char t13 = rt13 * (4 + t07);
        t13 += (1 - rt13) * t06; // 0, 1, 2, 3, 4, 5, 6, 7

        char rt20 = (input[id] > 7); // root 10
        char t20 = rt20 * (8 + t11);
        t20 += (1 - rt20) * t10;
        // 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15

        char rt21 = (input[id] > 23); // root 10
        char t21 = rt21 * (8 + t13);
        t21 += (1 - rt21) * t12;
        // 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15

        char root = (input[id] > 15); // root 10
        char leaf = root * (16 + t21);
        leaf += (1 - root) * t20;
        // 0, ... , 31

        vR[id] = leaf;
    }
}

#endif

// TODO
#if defined(_RF_WITH_CALC_1TREE_1INPUT_7LEVEL)

#define EXP_NAME "_RF_WITH_CALC_1TREE_1INPUT_7LEVEL"
#define EXP_OUTPUT_FILE "_RF_WITH_CALC_1TREE_1INPUT_7LEVEL.txt"
#define EXP_RESULT_FILE "_RF_WITH_CALC_1TREE_1INPUT_7LEVEL.csv"
#define _1_INPUT

// RF_with_calc
__global__ void RF_kernel(float *input, const int n, float *vR)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // para 128 classes
    if (id < n)
    {
        char rt0_00 = (input[id] > 1); // root t0_00
        char t0_00 = rt0_00 * (2 + (input[id] > 2));
        t0_00 += (1 - rt0_00) * (input[id] > 0); // 0, 1, 2, 3

        char rt0_01 = (input[id] > 5); // root t0_01
        char t0_01 = rt0_01 * (2 + (input[id] > 6));
        t0_01 += (1 - rt0_01) * (input[id] > 4); // 0, 1, 2, 3

        char rt0_02 = (input[id] > 9); // root t0_02
        char t0_02 = rt0_02 * (2 + (input[id] > 10));
        t0_02 += (1 - rt0_02) * (input[id] > 8); // 0, 1, 2, 3

        char rt0_03 = (input[id] > 13); // root t0_03
        char t0_03 = rt0_03 * (2 + (input[id] > 14));
        t0_03 += (1 - rt0_03) * (input[id] > 12); // 0, 1, 2, 3

        char rt0_04 = (input[id] > 17); // root t0_04
        char t0_04 = rt0_04 * (2 + (input[id] > 18));
        t0_04 += (1 - rt0_04) * (input[id] > 16); // 0, 1, 2, 3

        char rt0_05 = (input[id] > 21); // root t0_05
        char t0_05 = rt0_05 * (2 + (input[id] > 22));
        t0_05 += (1 - rt0_05) * (input[id] > 20); // 0, 1, 2, 3

        char rt0_06 = (input[id] > 25); // root t0_06
        char t0_06 = rt0_06 * (2 + (input[id] > 26));
        t0_06 += (1 - rt0_06) * (input[id] > 24); // 0, 1, 2, 3

        char rt0_07 = (input[id] > 29); // root t0_07
        char t0_07 = rt0_07 * (2 + (input[id] > 30));
        t0_07 += (1 - rt0_07) * (input[id] > 28); // 0, 1, 2, 3

        char rt0_08 = (input[id] > 33); // root t0_08
        char t0_08 = rt0_08 * (2 + (input[id] > 34));
        t0_08 += (1 - rt0_08) * (input[id] > 32); // 0, 1, 2, 3

        char rt0_09 = (input[id] > 37); // root t0_09
        char t0_09 = rt0_09 * (2 + (input[id] > 38));
        t0_09 += (1 - rt0_09) * (input[id] > 36); // 0, 1, 2, 3

        char rt0_10 = (input[id] > 41); // root t0_10
        char t0_10 = rt0_10 * (2 + (input[id] > 42));
        t0_10 += (1 - rt0_10) * (input[id] > 40); // 0, 1, 2, 3

        char rt0_11 = (input[id] > 45); // root t0_11
        char t0_11 = rt0_11 * (2 + (input[id] > 46));
        t0_11 += (1 - rt0_11) * (input[id] > 44); // 0, 1, 2, 3

        char rt0_12 = (input[id] > 49); // root t0_12
        char t0_12 = rt0_12 * (2 + (input[id] > 50));
        t0_12 += (1 - rt0_12) * (input[id] > 48); // 0, 1, 2, 3

        char rt0_13 = (input[id] > 53); // root t0_13
        char t0_13 = rt0_13 * (2 + (input[id] > 54));
        t0_13 += (1 - rt0_13) * (input[id] > 52); // 0, 1, 2, 3

        char rt0_14 = (input[id] > 57); // root t0_14
        char t0_14 = rt0_14 * (2 + (input[id] > 58));
        t0_14 += (1 - rt0_14) * (input[id] > 56); // 0, 1, 2, 3

        char rt0_15 = (input[id] > 61); // root t0_15
        char t0_15 = rt0_15 * (2 + (input[id] > 62));
        t0_15 += (1 - rt0_15) * (input[id] > 60); // 0, 1, 2, 3

        char rt0_16 = (input[id] > 65); // root t0_16
        char t0_16 = rt0_16 * (2 + (input[id] > 66));
        t0_16 += (1 - rt0_16) * (input[id] > 64); // 0, 1, 2, 3

        char rt0_17 = (input[id] > 69); // root t0_17
        char t0_17 = rt0_17 * (2 + (input[id] > 70));
        t0_17 += (1 - rt0_17) * (input[id] > 68); // 0, 1, 2, 3

        char rt0_18 = (input[id] > 73); // root t0_18
        char t0_18 = rt0_18 * (2 + (input[id] > 74));
        t0_18 += (1 - rt0_18) * (input[id] > 72); // 0, 1, 2, 3

        char rt0_19 = (input[id] > 77); // root t0_19
        char t0_19 = rt0_19 * (2 + (input[id] > 78));
        t0_19 += (1 - rt0_19) * (input[id] > 76); // 0, 1, 2, 3

        char rt0_20 = (input[id] > 81); // root t0_20
        char t0_20 = rt0_20 * (2 + (input[id] > 82));
        t0_20 += (1 - rt0_20) * (input[id] > 80); // 0, 1, 2, 3

        char rt0_21 = (input[id] > 85); // root t0_21
        char t0_21 = rt0_21 * (2 + (input[id] > 86));
        t0_21 += (1 - rt0_21) * (input[id] > 84); // 0, 1, 2, 3

        char rt0_22 = (input[id] > 89); // root t0_22
        char t0_22 = rt0_22 * (2 + (input[id] > 90));
        t0_22 += (1 - rt0_22) * (input[id] > 88); // 0, 1, 2, 3

        char rt0_23 = (input[id] > 93); // root t0_23
        char t0_23 = rt0_23 * (2 + (input[id] > 94));
        t0_23 += (1 - rt0_23) * (input[id] > 92); // 0, 1, 2, 3

        char rt0_24 = (input[id] > 97); // root t0_24
        char t0_24 = rt0_24 * (2 + (input[id] > 98));
        t0_24 += (1 - rt0_24) * (input[id] > 96); // 0, 1, 2, 3

        char rt0_25 = (input[id] > 101); // root t0_25
        char t0_25 = rt0_25 * (2 + (input[id] > 102));
        t0_25 += (1 - rt0_25) * (input[id] > 100); // 0, 1, 2, 3

        char rt0_26 = (input[id] > 105); // root t0_26
        char t0_26 = rt0_26 * (2 + (input[id] > 106));
        t0_26 += (1 - rt0_26) * (input[id] > 104); // 0, 1, 2, 3

        char rt0_27 = (input[id] > 109); // root t0_27
        char t0_27 = rt0_27 * (2 + (input[id] > 110));
        t0_27 += (1 - rt0_27) * (input[id] > 108); // 0, 1, 2, 3

        char rt0_28 = (input[id] > 113); // root t0_28
        char t0_28 = rt0_28 * (2 + (input[id] > 114));
        t0_28 += (1 - rt0_28) * (input[id] > 112); // 0, 1, 2, 3

        char rt0_29 = (input[id] > 117); // root t0_29
        char t0_29 = rt0_29 * (2 + (input[id] > 118));
        t0_29 += (1 - rt0_29) * (input[id] > 116); // 0, 1, 2, 3

        char rt0_30 = (input[id] > 121); // root t0_30
        char t0_30 = rt0_30 * (2 + (input[id] > 122));
        t0_30 += (1 - rt0_30) * (input[id] > 120); // 0, 1, 2, 3

        char rt0_31 = (input[id] > 125); // root t0_31
        char t0_31 = rt0_31 * (2 + (input[id] > 126));
        t0_31 += (1 - rt0_31) * (input[id] > 124); // 0, 1, 2, 3

        char rt1_00 = (input[id] > 3);     // root 1_00
        char t1_00 = rt1_00 * (4 + t0_01); // 1
        t1_00 += (1 - rt1_00) * t0_00;     // 0, 1, 2, 3, 4, 5, 6, 7

        char rt1_01 = (input[id] > 11);    // root 1_01
        char t1_01 = rt1_01 * (4 + t0_03); // 1
        t1_01 += (1 - rt1_01) * t0_02;     // 0, 1, 2, 3, 4, 5, 6, 7

        char rt1_02 = (input[id] > 19);    // root 1_02
        char t1_02 = rt1_02 * (4 + t0_05); // 1
        t1_02 += (1 - rt1_02) * t0_04;     // 0, 1, 2, 3, 4, 5, 6, 7

        char rt1_03 = (input[id] > 27);    // root 1_03
        char t1_03 = rt1_03 * (4 + t0_07); // 1
        t1_03 += (1 - rt1_03) * t0_06;     // 0, 1, 2, 3, 4, 5, 6, 7

        char rt1_04 = (input[id] > 35);    // root 1_04
        char t1_04 = rt1_04 * (4 + t0_09); // 1
        t1_04 += (1 - rt1_04) * t0_08;     // 0, 1, 2, 3, 4, 5, 6, 7

        char rt1_05 = (input[id] > 43);    // root 1_05
        char t1_05 = rt1_05 * (4 + t0_11); // 1
        t1_05 += (1 - rt1_05) * t0_10;     // 0, 1, 2, 3, 4, 5, 6, 7

        char rt1_06 = (input[id] > 51);    // root 1_06
        char t1_06 = rt1_06 * (4 + t0_13); // 1
        t1_06 += (1 - rt1_06) * t0_12;     // 0, 1, 2, 3, 4, 5, 6, 7

        char rt1_07 = (input[id] > 59);    // root 1_07
        char t1_07 = rt1_07 * (4 + t0_15); // 1
        t1_07 += (1 - rt1_07) * t0_14;     // 0, 1, 2, 3, 4, 5, 6, 7

        char rt1_08 = (input[id] > 67);    // root 1_08
        char t1_08 = rt1_08 * (4 + t0_17); // 1
        t1_08 += (1 - rt1_08) * t0_16;     // 0, 1, 2, 3, 4, 5, 6, 7

        char rt1_09 = (input[id] > 75);    // root 1_09
        char t1_09 = rt1_09 * (4 + t0_19); // 1
        t1_09 += (1 - rt1_09) * t0_18;     // 0, 1, 2, 3, 4, 5, 6, 7

        char rt1_10 = (input[id] > 83);    // root 1_10
        char t1_10 = rt1_10 * (4 + t0_21); // 1
        t1_10 += (1 - rt1_10) * t0_20;     // 0, 1, 2, 3, 4, 5, 6, 7

        char rt1_11 = (input[id] > 91);    // root 1_11
        char t1_11 = rt1_11 * (4 + t0_23); // 1
        t1_11 += (1 - rt1_11) * t0_22;     // 0, 1, 2, 3, 4, 5, 6, 7

        char rt1_12 = (input[id] > 99);    // root 1_12
        char t1_12 = rt1_12 * (4 + t0_25); // 1
        t1_12 += (1 - rt1_12) * t0_24;     // 0, 1, 2, 3, 4, 5, 6, 7

        char rt1_13 = (input[id] > 107);   // root 1_13
        char t1_13 = rt1_13 * (4 + t0_27); // 1
        t1_13 += (1 - rt1_13) * t0_26;     // 0, 1, 2, 3, 4, 5, 6, 7

        char rt1_14 = (input[id] > 115);   // root 1_14
        char t1_14 = rt1_14 * (4 + t0_29); // 1
        t1_14 += (1 - rt1_14) * t0_28;     // 0, 1, 2, 3, 4, 5, 6, 7

        char rt1_15 = (input[id] > 123);   // root 1_15
        char t1_15 = rt1_15 * (4 + t0_31); // 1
        t1_15 += (1 - rt1_15) * t0_30;     // 0, 1, 2, 3, 4, 5, 6, 7

        char rt2_00 = (input[id] > 7); // root 2_00
        char t2_00 = rt2_00 * (8 + t1_01);
        t2_00 += (1 - rt2_00) * t1_00;
        // 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15

        char rt2_01 = (input[id] > 23); // root 2_01
        char t2_01 = rt2_01 * (8 + t1_03);
        t2_01 += (1 - rt2_01) * t1_02;
        // 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15

        char rt2_02 = (input[id] > 39); // root 2_02
        char t2_02 = rt2_02 * (8 + t1_05);
        t2_02 += (1 - rt2_02) * t1_04;
        // 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15

        char rt2_03 = (input[id] > 55); // root 2_03
        char t2_03 = rt2_03 * (8 + t1_07);
        t2_03 += (1 - rt2_03) * t1_06;
        // 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15

        char rt2_04 = (input[id] > 71); // root 2_04
        char t2_04 = rt2_04 * (8 + t1_09);
        t2_04 += (1 - rt2_04) * t1_08;
        // 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15

        char rt2_05 = (input[id] > 87); // root 2_05
        char t2_05 = rt2_05 * (8 + t1_11);
        t2_05 += (1 - rt2_05) * t1_10;
        // 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15

        char rt2_06 = (input[id] > 103); // root 2_06
        char t2_06 = rt2_06 * (8 + t1_13);
        t2_06 += (1 - rt2_06) * t1_12;
        // 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15

        char rt2_07 = (input[id] > 119); // root 2_07
        char t2_07 = rt2_07 * (8 + t1_15);
        t2_07 += (1 - rt2_07) * t1_14;
        // 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15

        char rt3_00 = (input[id] > 15); // root 3_00
        char t3_00 = rt3_00 * (16 + t2_01);
        t3_00 += (1 - rt3_00) * t2_00;
        // 0, ..., 31

        char rt3_01 = (input[id] > 47); // root 3_01
        char t3_01 = rt3_01 * (16 + t2_03);
        t3_01 += (1 - rt3_01) * t2_02;
        // 0, ..., 31

        char rt3_02 = (input[id] > 79); // root 3_02
        char t3_02 = rt3_02 * (16 + t2_05);
        t3_02 += (1 - rt3_02) * t2_04;
        // 0, ..., 31

        char rt3_03 = (input[id] > 111); // root 3_03
        char t3_03 = rt3_03 * (16 + t2_07);
        t3_03 += (1 - rt3_03) * t2_06;
        // 0, ..., 31

        char rt4_00 = (input[id] > 31); // root 4_00
        char t4_00 = rt4_00 * (32 + t3_01);
        t4_00 += (1 - rt4_00) * t3_00;
        // 0, ..., 64

        char rt4_01 = (input[id] > 95); // root 4_01
        char t4_01 = rt4_01 * (32 + t3_03);
        t4_01 += (1 - rt4_01) * t3_02;
        // 0, ..., 64

        char root = (input[id] > 63); // root 10
        char leaf = root * (64 + t4_01);
        leaf += (1 - root) * t4_00;
        // 0, ... , 128

        vR[id] = leaf;
    }
}

#endif
//*********

void initData(float *ip, int size);
void writeOutFile(int value);
void registerTime(float value);

int main(int argc, char **argv)
{

    float elapsed_time;
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("[CUDA - Random Forest]: EXP %s\n", EXP_NAME);
    printf("[CUDA - Random Forest]: Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set up data size of vectors
    int nElem = N_ELEM;
    printf("[CUDA - Random Forest]: Vector Size %d\n", nElem);

    // malloc host memory
    size_t nBytes = nElem * sizeof(float);

    float *h_vR;
    h_vR = (float *)malloc(nBytes);

#if defined(_1_INPUT)
    float *h_input;
    h_input = (float *)malloc(nBytes);
#endif

    printf("[CUDA - Random Forest]: Start Reading Dataset\n");
#if defined(_1_INPUT)
    initData(h_input, nElem);
#endif
    printf("[CUDA - Random Forest]: Finished Reading Dataset\n");

    memset(h_vR, 0, nBytes);

    // malloc device global memory
    float *d_vR;
    CHECK(hipMalloc((float **)&d_vR, nBytes));

#if defined(_1_INPUT)
    float *d_input;
    CHECK(hipMalloc((float **)&d_input, nBytes));
#endif

    // transfer data from host to device
#if defined(_1_INPUT)
    CHECK(hipMemcpy(d_input, h_input, nBytes, hipMemcpyHostToDevice));
#endif

    // invoke kernel at host side
    int iLen = 512;
    dim3 block(iLen);
    dim3 grid((nElem + block.x - 1) / block.x);

    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));
    // record start event
    CHECK(hipEventRecord(start, 0));

#if defined(_RF_WITH_CALC_1TREE_1INPUT_5LEVEL) || defined(_RF_WITH_IF_1TREE_1INPUT_7LEVEL_BEST) ||     \
    defined(_RF_WITH_IF_1TREE_1INPUT_7LEVEL_WORST) || defined(_RF_WITH_IF_4TREE_1INPUT_5LEVEL_BEST) || \
    defined(_RF_WITH_IF_4TREE_1INPUT_5LEVEL_WORST)
    RF_kernel<<<grid, block>>>(d_input, nElem, d_vR);
#endif

    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    hipDeviceSynchronize();
    // calculate elapsed time
    CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
    printf("[CUDA - Random Forest]: %s - execution time = %.6fms\n", EXP_NAME, elapsed_time);

    registerTime(elapsed_time);

    // copy kernel result back to host side
#if defined(_1_INPUT)
    CHECK(hipMemcpy(h_vR, d_vR, nBytes, hipMemcpyDeviceToHost));
    CHECK(hipGetLastError());
#endif

    char flag = 1;

    for (int i = 0; i < nElem; i++)
    {
        if (i < 32)
        {
            printf("%.3f ", h_vR[i]);
        }
#if defined(_1_INPUT)
        if (h_input[i] != h_vR[i])
        {
            printf("Error!!");
            flag = 0;
            break;
        }
#endif
    }

    if (flag == 1)
    {
        printf("Return correct");
    }

    printf("\n ");

    // free device memory
#if defined(_1_INPUT)
    hipFree(d_input);
#endif
    hipFree(d_vR);

    // free host memory
#if defined(_1_INPUT)
    free(h_input);
#endif
    free(h_vR);

    return (0);
}

void initData(float *ip, int size)
{
    for (int i = 0; i < size; i++)
    {
#if defined(_RF_WITH_IF_1TREE_1INPUT_7LEVEL_WORST)
        ip[i] = (float)((i % WARP_SIZE) * 4);

#elif defined(_RF_WITH_IF_1TREE_1INPUT_3LEVEL_BEST) || defined(_RF_WITH_IF_1TREE_1INPUT_7LEVEL_BEST) || \
    defined(_RF_WITH_IF_4TREE_1INPUT_5LEVEL_BEST)
        ip[i] = (float)(0);

#elif defined(_RF_WITH_IF_1TREE_1INPUT_3LEVEL_WORST) || defined(_RF_WITH_IF_4TREE_1INPUT_5LEVEL_WORST)
        ip[i] = (float)(i % WARP_SIZE);
#endif
    }
    return;
}

void writeOutFile(int value)
{
    outFile = fopen(EXP_OUTPUT_FILE, "a");
    fprintf(outFile, "%d\n", value);
    fclose(outFile);
}

void registerTime(float value)
{
    printf("[CUDA - Random Forest]: Time spent: %f.\n", value);
    outFile = fopen(EXP_RESULT_FILE, "a");
    fprintf(outFile, ",%f", value);
    fclose(outFile);
}
