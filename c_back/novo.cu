#include "hip/hip_runtime.h"
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>

#define WARP_SIZE 32
#define N_ELEM 200000000

#define CHECK(call)                                                \
    {                                                              \
        const hipError_t error = call;                            \
        if (error != hipSuccess)                                  \
        {                                                          \
            fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
            fprintf(stderr, "code: %d, reason: %s\n", error,       \
                    hipGetErrorString(error));                    \
        }                                                          \
    }
FILE *outFile;

// RF_kernel
__global__ void RF_copia_simples(float *input, const int n, float *vR)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // para 8 classes
    if (i < n)
    {
        vR[i] = input[i];
    }
}

// RF_kernel
__global__ void RF_1if(float *input, const int n, float *vR)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // para 8 classes
    if (i < n)
    {
        if (input[i] > 16)
        {
            vR[i] = 17;
        }
        else
        {
            vR[i] = 15;
        }
    }
}

// RF_kernel
__global__ void RF_1if2t(float *input, const int n, const int offset, float *vR)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int local;
    // para 8 classes
    if (i < n)
    {
        if (input[i] > 16)
        {
            local = 17;
        }
        else
        {
            local = 15;
        }

        if (input[i] > 16 + offset)
        {
            vR[i] = local;
        }
        else
        {
            vR[i] = local;
        }
    }
}

// RF_kernel
__global__ void RF_2if(float *input, const int n, float *vR)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // para 8 classes
    if (i < n)
    {
        if (input[i] > 16)
        {
            if (input[i] > 17)
            {
                vR[i] = 18;
            }
            else
            {
                vR[i] = 17;
            }
        }
        else
        {
            vR[i] = 15;
        }
    }
}

int main(int argc, char **argv)
{

    float elapsed_time;
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("[CUDA - Random Forest]: EXP %s\n", EXP_NAME);
    printf("[CUDA - Random Forest]: Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set up data size of vectors
    int nElem = N_ELEM;
    printf("[CUDA - Random Forest]: Vector Size %d\n", nElem);

    // malloc host memory
    size_t nBytes = nElem * sizeof(float);

    float *h_vR;
    h_vR = (float *)malloc(nBytes);

#if defined(_1_INPUT)
    float *h_input;
    h_input = (float *)malloc(nBytes);
#endif

    printf("[CUDA - Random Forest]: Start Reading Dataset\n");
#if defined(_1_INPUT)
    initData(h_input, nElem);
#endif
    printf("[CUDA - Random Forest]: Finished Reading Dataset\n");

    memset(h_vR, 0, nBytes);

    // malloc device global memory
    float *d_vR;
    CHECK(hipMalloc((float **)&d_vR, nBytes));

#if defined(_1_INPUT)
    float *d_input;
    CHECK(hipMalloc((float **)&d_input, nBytes));
#endif

    // transfer data from host to device
#if defined(_1_INPUT)
    CHECK(hipMemcpy(d_input, h_input, nBytes, hipMemcpyHostToDevice));
#endif

    // invoke kernel at host side
    int iLen = 512;
    dim3 block(iLen);
    dim3 grid((nElem + block.x - 1) / block.x);

    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));
    // record start event
    CHECK(hipEventRecord(start, 0));

#if defined(_RF_WITH_CALC_1TREE_1INPUT_5LEVEL) || defined(_RF_WITH_IF_1TREE_1INPUT_7LEVEL_BEST) ||     \
    defined(_RF_WITH_IF_1TREE_1INPUT_7LEVEL_WORST) || defined(_RF_WITH_IF_4TREE_1INPUT_5LEVEL_BEST) || \
    defined(_RF_WITH_IF_4TREE_1INPUT_5LEVEL_WORST)
    RF_kernel<<<grid, block>>>(d_input, nElem, d_vR);
#endif

    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    hipDeviceSynchronize();
    // calculate elapsed time
    CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
    printf("[CUDA - Random Forest]: %s - execution time = %.6fms\n", EXP_NAME, elapsed_time);

    registerTime(elapsed_time);

    // copy kernel result back to host side
#if defined(_1_INPUT)
    CHECK(hipMemcpy(h_vR, d_vR, nBytes, hipMemcpyDeviceToHost));
    CHECK(hipGetLastError());
#endif

    char flag = 1;

    for (int i = 0; i < nElem; i++)
    {
        if (i < 32)
        {
            printf("%.3f ", h_vR[i]);
        }
#if defined(_1_INPUT)
        if (h_input[i] != h_vR[i])
        {
            printf("Error!!");
            flag = 0;
            break;
        }
#endif
    }

    if (flag == 1)
    {
        printf("Return correct");
    }

    printf("\n ");

    // free device memory
#if defined(_1_INPUT)
    hipFree(d_input);
#endif
    hipFree(d_vR);

    // free host memory
#if defined(_1_INPUT)
    free(h_input);
#endif
    free(h_vR);

    return (0);
}

void initData(float *ip, int size)
{
    for (int i = 0; i < size; i++)
    {
#if defined(_RF_WITH_IF_1TREE_1INPUT_7LEVEL_WORST)
        ip[i] = (float)((i % WARP_SIZE) * 4);

#elif defined(_RF_WITH_IF_1TREE_1INPUT_3LEVEL_BEST) || defined(_RF_WITH_IF_1TREE_1INPUT_7LEVEL_BEST) || \
    defined(_RF_WITH_IF_4TREE_1INPUT_5LEVEL_BEST)
        ip[i] = (float)(0);

#elif defined(_RF_WITH_IF_1TREE_1INPUT_3LEVEL_WORST) || defined(_RF_WITH_IF_4TREE_1INPUT_5LEVEL_WORST)
        ip[i] = (float)(i % WARP_SIZE);
#endif
    }
    return;
}

void writeOutFile(int value)
{
    outFile = fopen(EXP_OUTPUT_FILE, "a");
    fprintf(outFile, "%d\n", value);
    fclose(outFile);
}

void registerTime(float value)
{
    printf("[CUDA - Random Forest]: Time spent: %f.\n", value);
    outFile = fopen(EXP_RESULT_FILE, "a");
    fprintf(outFile, ",%f", value);
    fclose(outFile);
}
