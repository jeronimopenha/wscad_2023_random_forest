#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>

#define _WORST
#define TIMES 10
#define N_ELEM 100000000

#define WARP_SIZE 32

#define CHECK(call)                                                \
	{                                                              \
		const hipError_t error = call;                            \
		if (error != hipSuccess)                                  \
		{                                                          \
			fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
			fprintf(stderr, "code: %d, reason: %s\n", error,       \
					hipGetErrorString(error));                    \
		}                                                          \
	}

void initData(float *ip, int size);
void registerTime(float value);

#define _MOD 2048
#define _4TREE
#define EXP_NAME "_RF_2047IF_4TREE_1INPUT__WORST_100000000"

// RF_with_if

__global__ void RF_kernel(float *input, const int n, const int offset0, const int offset1, const int offset2, const int offset3, float *vR)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
if(input[i]>(1023+offset0))
{
	if(input[i]>(1535+offset0))
{
	if(input[i]>(1791+offset0))
{
	if(input[i]>(1919+offset0))
{
	if(input[i]>(1983+offset0))
{
	if(input[i]>(2015+offset0))
{
	if(input[i]>(2031+offset0))
{
	if(input[i]>(2039+offset0))
{
	if(input[i]>(2043+offset0))
{
	if(input[i]>(2045+offset0))
{
	if(input[i]==(2047+offset0))
{
	vR[i] = 2047;
}
else
{
	vR[i] = 2046;
}

}
else
{
	if(input[i]==(2045+offset0))
{
	vR[i] = 2045;
}
else
{
	vR[i] = 2044;
}

}

}
else
{
	if(input[i]>(2041+offset0))
{
	if(input[i]==(2043+offset0))
{
	vR[i] = 2043;
}
else
{
	vR[i] = 2042;
}

}
else
{
	if(input[i]==(2041+offset0))
{
	vR[i] = 2041;
}
else
{
	vR[i] = 2040;
}

}

}

}
else
{
	if(input[i]>(2035+offset0))
{
	if(input[i]>(2037+offset0))
{
	if(input[i]==(2039+offset0))
{
	vR[i] = 2039;
}
else
{
	vR[i] = 2038;
}

}
else
{
	if(input[i]==(2037+offset0))
{
	vR[i] = 2037;
}
else
{
	vR[i] = 2036;
}

}

}
else
{
	if(input[i]>(2033+offset0))
{
	if(input[i]==(2035+offset0))
{
	vR[i] = 2035;
}
else
{
	vR[i] = 2034;
}

}
else
{
	if(input[i]==(2033+offset0))
{
	vR[i] = 2033;
}
else
{
	vR[i] = 2032;
}

}

}

}

}
else
{
	if(input[i]>(2023+offset0))
{
	if(input[i]>(2027+offset0))
{
	if(input[i]>(2029+offset0))
{
	if(input[i]==(2031+offset0))
{
	vR[i] = 2031;
}
else
{
	vR[i] = 2030;
}

}
else
{
	if(input[i]==(2029+offset0))
{
	vR[i] = 2029;
}
else
{
	vR[i] = 2028;
}

}

}
else
{
	if(input[i]>(2025+offset0))
{
	if(input[i]==(2027+offset0))
{
	vR[i] = 2027;
}
else
{
	vR[i] = 2026;
}

}
else
{
	if(input[i]==(2025+offset0))
{
	vR[i] = 2025;
}
else
{
	vR[i] = 2024;
}

}

}

}
else
{
	if(input[i]>(2019+offset0))
{
	if(input[i]>(2021+offset0))
{
	if(input[i]==(2023+offset0))
{
	vR[i] = 2023;
}
else
{
	vR[i] = 2022;
}

}
else
{
	if(input[i]==(2021+offset0))
{
	vR[i] = 2021;
}
else
{
	vR[i] = 2020;
}

}

}
else
{
	if(input[i]>(2017+offset0))
{
	if(input[i]==(2019+offset0))
{
	vR[i] = 2019;
}
else
{
	vR[i] = 2018;
}

}
else
{
	if(input[i]==(2017+offset0))
{
	vR[i] = 2017;
}
else
{
	vR[i] = 2016;
}

}

}

}

}

}
else
{
	if(input[i]>(1999+offset0))
{
	if(input[i]>(2007+offset0))
{
	if(input[i]>(2011+offset0))
{
	if(input[i]>(2013+offset0))
{
	if(input[i]==(2015+offset0))
{
	vR[i] = 2015;
}
else
{
	vR[i] = 2014;
}

}
else
{
	if(input[i]==(2013+offset0))
{
	vR[i] = 2013;
}
else
{
	vR[i] = 2012;
}

}

}
else
{
	if(input[i]>(2009+offset0))
{
	if(input[i]==(2011+offset0))
{
	vR[i] = 2011;
}
else
{
	vR[i] = 2010;
}

}
else
{
	if(input[i]==(2009+offset0))
{
	vR[i] = 2009;
}
else
{
	vR[i] = 2008;
}

}

}

}
else
{
	if(input[i]>(2003+offset0))
{
	if(input[i]>(2005+offset0))
{
	if(input[i]==(2007+offset0))
{
	vR[i] = 2007;
}
else
{
	vR[i] = 2006;
}

}
else
{
	if(input[i]==(2005+offset0))
{
	vR[i] = 2005;
}
else
{
	vR[i] = 2004;
}

}

}
else
{
	if(input[i]>(2001+offset0))
{
	if(input[i]==(2003+offset0))
{
	vR[i] = 2003;
}
else
{
	vR[i] = 2002;
}

}
else
{
	if(input[i]==(2001+offset0))
{
	vR[i] = 2001;
}
else
{
	vR[i] = 2000;
}

}

}

}

}
else
{
	if(input[i]>(1991+offset0))
{
	if(input[i]>(1995+offset0))
{
	if(input[i]>(1997+offset0))
{
	if(input[i]==(1999+offset0))
{
	vR[i] = 1999;
}
else
{
	vR[i] = 1998;
}

}
else
{
	if(input[i]==(1997+offset0))
{
	vR[i] = 1997;
}
else
{
	vR[i] = 1996;
}

}

}
else
{
	if(input[i]>(1993+offset0))
{
	if(input[i]==(1995+offset0))
{
	vR[i] = 1995;
}
else
{
	vR[i] = 1994;
}

}
else
{
	if(input[i]==(1993+offset0))
{
	vR[i] = 1993;
}
else
{
	vR[i] = 1992;
}

}

}

}
else
{
	if(input[i]>(1987+offset0))
{
	if(input[i]>(1989+offset0))
{
	if(input[i]==(1991+offset0))
{
	vR[i] = 1991;
}
else
{
	vR[i] = 1990;
}

}
else
{
	if(input[i]==(1989+offset0))
{
	vR[i] = 1989;
}
else
{
	vR[i] = 1988;
}

}

}
else
{
	if(input[i]>(1985+offset0))
{
	if(input[i]==(1987+offset0))
{
	vR[i] = 1987;
}
else
{
	vR[i] = 1986;
}

}
else
{
	if(input[i]==(1985+offset0))
{
	vR[i] = 1985;
}
else
{
	vR[i] = 1984;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1951+offset0))
{
	if(input[i]>(1967+offset0))
{
	if(input[i]>(1975+offset0))
{
	if(input[i]>(1979+offset0))
{
	if(input[i]>(1981+offset0))
{
	if(input[i]==(1983+offset0))
{
	vR[i] = 1983;
}
else
{
	vR[i] = 1982;
}

}
else
{
	if(input[i]==(1981+offset0))
{
	vR[i] = 1981;
}
else
{
	vR[i] = 1980;
}

}

}
else
{
	if(input[i]>(1977+offset0))
{
	if(input[i]==(1979+offset0))
{
	vR[i] = 1979;
}
else
{
	vR[i] = 1978;
}

}
else
{
	if(input[i]==(1977+offset0))
{
	vR[i] = 1977;
}
else
{
	vR[i] = 1976;
}

}

}

}
else
{
	if(input[i]>(1971+offset0))
{
	if(input[i]>(1973+offset0))
{
	if(input[i]==(1975+offset0))
{
	vR[i] = 1975;
}
else
{
	vR[i] = 1974;
}

}
else
{
	if(input[i]==(1973+offset0))
{
	vR[i] = 1973;
}
else
{
	vR[i] = 1972;
}

}

}
else
{
	if(input[i]>(1969+offset0))
{
	if(input[i]==(1971+offset0))
{
	vR[i] = 1971;
}
else
{
	vR[i] = 1970;
}

}
else
{
	if(input[i]==(1969+offset0))
{
	vR[i] = 1969;
}
else
{
	vR[i] = 1968;
}

}

}

}

}
else
{
	if(input[i]>(1959+offset0))
{
	if(input[i]>(1963+offset0))
{
	if(input[i]>(1965+offset0))
{
	if(input[i]==(1967+offset0))
{
	vR[i] = 1967;
}
else
{
	vR[i] = 1966;
}

}
else
{
	if(input[i]==(1965+offset0))
{
	vR[i] = 1965;
}
else
{
	vR[i] = 1964;
}

}

}
else
{
	if(input[i]>(1961+offset0))
{
	if(input[i]==(1963+offset0))
{
	vR[i] = 1963;
}
else
{
	vR[i] = 1962;
}

}
else
{
	if(input[i]==(1961+offset0))
{
	vR[i] = 1961;
}
else
{
	vR[i] = 1960;
}

}

}

}
else
{
	if(input[i]>(1955+offset0))
{
	if(input[i]>(1957+offset0))
{
	if(input[i]==(1959+offset0))
{
	vR[i] = 1959;
}
else
{
	vR[i] = 1958;
}

}
else
{
	if(input[i]==(1957+offset0))
{
	vR[i] = 1957;
}
else
{
	vR[i] = 1956;
}

}

}
else
{
	if(input[i]>(1953+offset0))
{
	if(input[i]==(1955+offset0))
{
	vR[i] = 1955;
}
else
{
	vR[i] = 1954;
}

}
else
{
	if(input[i]==(1953+offset0))
{
	vR[i] = 1953;
}
else
{
	vR[i] = 1952;
}

}

}

}

}

}
else
{
	if(input[i]>(1935+offset0))
{
	if(input[i]>(1943+offset0))
{
	if(input[i]>(1947+offset0))
{
	if(input[i]>(1949+offset0))
{
	if(input[i]==(1951+offset0))
{
	vR[i] = 1951;
}
else
{
	vR[i] = 1950;
}

}
else
{
	if(input[i]==(1949+offset0))
{
	vR[i] = 1949;
}
else
{
	vR[i] = 1948;
}

}

}
else
{
	if(input[i]>(1945+offset0))
{
	if(input[i]==(1947+offset0))
{
	vR[i] = 1947;
}
else
{
	vR[i] = 1946;
}

}
else
{
	if(input[i]==(1945+offset0))
{
	vR[i] = 1945;
}
else
{
	vR[i] = 1944;
}

}

}

}
else
{
	if(input[i]>(1939+offset0))
{
	if(input[i]>(1941+offset0))
{
	if(input[i]==(1943+offset0))
{
	vR[i] = 1943;
}
else
{
	vR[i] = 1942;
}

}
else
{
	if(input[i]==(1941+offset0))
{
	vR[i] = 1941;
}
else
{
	vR[i] = 1940;
}

}

}
else
{
	if(input[i]>(1937+offset0))
{
	if(input[i]==(1939+offset0))
{
	vR[i] = 1939;
}
else
{
	vR[i] = 1938;
}

}
else
{
	if(input[i]==(1937+offset0))
{
	vR[i] = 1937;
}
else
{
	vR[i] = 1936;
}

}

}

}

}
else
{
	if(input[i]>(1927+offset0))
{
	if(input[i]>(1931+offset0))
{
	if(input[i]>(1933+offset0))
{
	if(input[i]==(1935+offset0))
{
	vR[i] = 1935;
}
else
{
	vR[i] = 1934;
}

}
else
{
	if(input[i]==(1933+offset0))
{
	vR[i] = 1933;
}
else
{
	vR[i] = 1932;
}

}

}
else
{
	if(input[i]>(1929+offset0))
{
	if(input[i]==(1931+offset0))
{
	vR[i] = 1931;
}
else
{
	vR[i] = 1930;
}

}
else
{
	if(input[i]==(1929+offset0))
{
	vR[i] = 1929;
}
else
{
	vR[i] = 1928;
}

}

}

}
else
{
	if(input[i]>(1923+offset0))
{
	if(input[i]>(1925+offset0))
{
	if(input[i]==(1927+offset0))
{
	vR[i] = 1927;
}
else
{
	vR[i] = 1926;
}

}
else
{
	if(input[i]==(1925+offset0))
{
	vR[i] = 1925;
}
else
{
	vR[i] = 1924;
}

}

}
else
{
	if(input[i]>(1921+offset0))
{
	if(input[i]==(1923+offset0))
{
	vR[i] = 1923;
}
else
{
	vR[i] = 1922;
}

}
else
{
	if(input[i]==(1921+offset0))
{
	vR[i] = 1921;
}
else
{
	vR[i] = 1920;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(1855+offset0))
{
	if(input[i]>(1887+offset0))
{
	if(input[i]>(1903+offset0))
{
	if(input[i]>(1911+offset0))
{
	if(input[i]>(1915+offset0))
{
	if(input[i]>(1917+offset0))
{
	if(input[i]==(1919+offset0))
{
	vR[i] = 1919;
}
else
{
	vR[i] = 1918;
}

}
else
{
	if(input[i]==(1917+offset0))
{
	vR[i] = 1917;
}
else
{
	vR[i] = 1916;
}

}

}
else
{
	if(input[i]>(1913+offset0))
{
	if(input[i]==(1915+offset0))
{
	vR[i] = 1915;
}
else
{
	vR[i] = 1914;
}

}
else
{
	if(input[i]==(1913+offset0))
{
	vR[i] = 1913;
}
else
{
	vR[i] = 1912;
}

}

}

}
else
{
	if(input[i]>(1907+offset0))
{
	if(input[i]>(1909+offset0))
{
	if(input[i]==(1911+offset0))
{
	vR[i] = 1911;
}
else
{
	vR[i] = 1910;
}

}
else
{
	if(input[i]==(1909+offset0))
{
	vR[i] = 1909;
}
else
{
	vR[i] = 1908;
}

}

}
else
{
	if(input[i]>(1905+offset0))
{
	if(input[i]==(1907+offset0))
{
	vR[i] = 1907;
}
else
{
	vR[i] = 1906;
}

}
else
{
	if(input[i]==(1905+offset0))
{
	vR[i] = 1905;
}
else
{
	vR[i] = 1904;
}

}

}

}

}
else
{
	if(input[i]>(1895+offset0))
{
	if(input[i]>(1899+offset0))
{
	if(input[i]>(1901+offset0))
{
	if(input[i]==(1903+offset0))
{
	vR[i] = 1903;
}
else
{
	vR[i] = 1902;
}

}
else
{
	if(input[i]==(1901+offset0))
{
	vR[i] = 1901;
}
else
{
	vR[i] = 1900;
}

}

}
else
{
	if(input[i]>(1897+offset0))
{
	if(input[i]==(1899+offset0))
{
	vR[i] = 1899;
}
else
{
	vR[i] = 1898;
}

}
else
{
	if(input[i]==(1897+offset0))
{
	vR[i] = 1897;
}
else
{
	vR[i] = 1896;
}

}

}

}
else
{
	if(input[i]>(1891+offset0))
{
	if(input[i]>(1893+offset0))
{
	if(input[i]==(1895+offset0))
{
	vR[i] = 1895;
}
else
{
	vR[i] = 1894;
}

}
else
{
	if(input[i]==(1893+offset0))
{
	vR[i] = 1893;
}
else
{
	vR[i] = 1892;
}

}

}
else
{
	if(input[i]>(1889+offset0))
{
	if(input[i]==(1891+offset0))
{
	vR[i] = 1891;
}
else
{
	vR[i] = 1890;
}

}
else
{
	if(input[i]==(1889+offset0))
{
	vR[i] = 1889;
}
else
{
	vR[i] = 1888;
}

}

}

}

}

}
else
{
	if(input[i]>(1871+offset0))
{
	if(input[i]>(1879+offset0))
{
	if(input[i]>(1883+offset0))
{
	if(input[i]>(1885+offset0))
{
	if(input[i]==(1887+offset0))
{
	vR[i] = 1887;
}
else
{
	vR[i] = 1886;
}

}
else
{
	if(input[i]==(1885+offset0))
{
	vR[i] = 1885;
}
else
{
	vR[i] = 1884;
}

}

}
else
{
	if(input[i]>(1881+offset0))
{
	if(input[i]==(1883+offset0))
{
	vR[i] = 1883;
}
else
{
	vR[i] = 1882;
}

}
else
{
	if(input[i]==(1881+offset0))
{
	vR[i] = 1881;
}
else
{
	vR[i] = 1880;
}

}

}

}
else
{
	if(input[i]>(1875+offset0))
{
	if(input[i]>(1877+offset0))
{
	if(input[i]==(1879+offset0))
{
	vR[i] = 1879;
}
else
{
	vR[i] = 1878;
}

}
else
{
	if(input[i]==(1877+offset0))
{
	vR[i] = 1877;
}
else
{
	vR[i] = 1876;
}

}

}
else
{
	if(input[i]>(1873+offset0))
{
	if(input[i]==(1875+offset0))
{
	vR[i] = 1875;
}
else
{
	vR[i] = 1874;
}

}
else
{
	if(input[i]==(1873+offset0))
{
	vR[i] = 1873;
}
else
{
	vR[i] = 1872;
}

}

}

}

}
else
{
	if(input[i]>(1863+offset0))
{
	if(input[i]>(1867+offset0))
{
	if(input[i]>(1869+offset0))
{
	if(input[i]==(1871+offset0))
{
	vR[i] = 1871;
}
else
{
	vR[i] = 1870;
}

}
else
{
	if(input[i]==(1869+offset0))
{
	vR[i] = 1869;
}
else
{
	vR[i] = 1868;
}

}

}
else
{
	if(input[i]>(1865+offset0))
{
	if(input[i]==(1867+offset0))
{
	vR[i] = 1867;
}
else
{
	vR[i] = 1866;
}

}
else
{
	if(input[i]==(1865+offset0))
{
	vR[i] = 1865;
}
else
{
	vR[i] = 1864;
}

}

}

}
else
{
	if(input[i]>(1859+offset0))
{
	if(input[i]>(1861+offset0))
{
	if(input[i]==(1863+offset0))
{
	vR[i] = 1863;
}
else
{
	vR[i] = 1862;
}

}
else
{
	if(input[i]==(1861+offset0))
{
	vR[i] = 1861;
}
else
{
	vR[i] = 1860;
}

}

}
else
{
	if(input[i]>(1857+offset0))
{
	if(input[i]==(1859+offset0))
{
	vR[i] = 1859;
}
else
{
	vR[i] = 1858;
}

}
else
{
	if(input[i]==(1857+offset0))
{
	vR[i] = 1857;
}
else
{
	vR[i] = 1856;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1823+offset0))
{
	if(input[i]>(1839+offset0))
{
	if(input[i]>(1847+offset0))
{
	if(input[i]>(1851+offset0))
{
	if(input[i]>(1853+offset0))
{
	if(input[i]==(1855+offset0))
{
	vR[i] = 1855;
}
else
{
	vR[i] = 1854;
}

}
else
{
	if(input[i]==(1853+offset0))
{
	vR[i] = 1853;
}
else
{
	vR[i] = 1852;
}

}

}
else
{
	if(input[i]>(1849+offset0))
{
	if(input[i]==(1851+offset0))
{
	vR[i] = 1851;
}
else
{
	vR[i] = 1850;
}

}
else
{
	if(input[i]==(1849+offset0))
{
	vR[i] = 1849;
}
else
{
	vR[i] = 1848;
}

}

}

}
else
{
	if(input[i]>(1843+offset0))
{
	if(input[i]>(1845+offset0))
{
	if(input[i]==(1847+offset0))
{
	vR[i] = 1847;
}
else
{
	vR[i] = 1846;
}

}
else
{
	if(input[i]==(1845+offset0))
{
	vR[i] = 1845;
}
else
{
	vR[i] = 1844;
}

}

}
else
{
	if(input[i]>(1841+offset0))
{
	if(input[i]==(1843+offset0))
{
	vR[i] = 1843;
}
else
{
	vR[i] = 1842;
}

}
else
{
	if(input[i]==(1841+offset0))
{
	vR[i] = 1841;
}
else
{
	vR[i] = 1840;
}

}

}

}

}
else
{
	if(input[i]>(1831+offset0))
{
	if(input[i]>(1835+offset0))
{
	if(input[i]>(1837+offset0))
{
	if(input[i]==(1839+offset0))
{
	vR[i] = 1839;
}
else
{
	vR[i] = 1838;
}

}
else
{
	if(input[i]==(1837+offset0))
{
	vR[i] = 1837;
}
else
{
	vR[i] = 1836;
}

}

}
else
{
	if(input[i]>(1833+offset0))
{
	if(input[i]==(1835+offset0))
{
	vR[i] = 1835;
}
else
{
	vR[i] = 1834;
}

}
else
{
	if(input[i]==(1833+offset0))
{
	vR[i] = 1833;
}
else
{
	vR[i] = 1832;
}

}

}

}
else
{
	if(input[i]>(1827+offset0))
{
	if(input[i]>(1829+offset0))
{
	if(input[i]==(1831+offset0))
{
	vR[i] = 1831;
}
else
{
	vR[i] = 1830;
}

}
else
{
	if(input[i]==(1829+offset0))
{
	vR[i] = 1829;
}
else
{
	vR[i] = 1828;
}

}

}
else
{
	if(input[i]>(1825+offset0))
{
	if(input[i]==(1827+offset0))
{
	vR[i] = 1827;
}
else
{
	vR[i] = 1826;
}

}
else
{
	if(input[i]==(1825+offset0))
{
	vR[i] = 1825;
}
else
{
	vR[i] = 1824;
}

}

}

}

}

}
else
{
	if(input[i]>(1807+offset0))
{
	if(input[i]>(1815+offset0))
{
	if(input[i]>(1819+offset0))
{
	if(input[i]>(1821+offset0))
{
	if(input[i]==(1823+offset0))
{
	vR[i] = 1823;
}
else
{
	vR[i] = 1822;
}

}
else
{
	if(input[i]==(1821+offset0))
{
	vR[i] = 1821;
}
else
{
	vR[i] = 1820;
}

}

}
else
{
	if(input[i]>(1817+offset0))
{
	if(input[i]==(1819+offset0))
{
	vR[i] = 1819;
}
else
{
	vR[i] = 1818;
}

}
else
{
	if(input[i]==(1817+offset0))
{
	vR[i] = 1817;
}
else
{
	vR[i] = 1816;
}

}

}

}
else
{
	if(input[i]>(1811+offset0))
{
	if(input[i]>(1813+offset0))
{
	if(input[i]==(1815+offset0))
{
	vR[i] = 1815;
}
else
{
	vR[i] = 1814;
}

}
else
{
	if(input[i]==(1813+offset0))
{
	vR[i] = 1813;
}
else
{
	vR[i] = 1812;
}

}

}
else
{
	if(input[i]>(1809+offset0))
{
	if(input[i]==(1811+offset0))
{
	vR[i] = 1811;
}
else
{
	vR[i] = 1810;
}

}
else
{
	if(input[i]==(1809+offset0))
{
	vR[i] = 1809;
}
else
{
	vR[i] = 1808;
}

}

}

}

}
else
{
	if(input[i]>(1799+offset0))
{
	if(input[i]>(1803+offset0))
{
	if(input[i]>(1805+offset0))
{
	if(input[i]==(1807+offset0))
{
	vR[i] = 1807;
}
else
{
	vR[i] = 1806;
}

}
else
{
	if(input[i]==(1805+offset0))
{
	vR[i] = 1805;
}
else
{
	vR[i] = 1804;
}

}

}
else
{
	if(input[i]>(1801+offset0))
{
	if(input[i]==(1803+offset0))
{
	vR[i] = 1803;
}
else
{
	vR[i] = 1802;
}

}
else
{
	if(input[i]==(1801+offset0))
{
	vR[i] = 1801;
}
else
{
	vR[i] = 1800;
}

}

}

}
else
{
	if(input[i]>(1795+offset0))
{
	if(input[i]>(1797+offset0))
{
	if(input[i]==(1799+offset0))
{
	vR[i] = 1799;
}
else
{
	vR[i] = 1798;
}

}
else
{
	if(input[i]==(1797+offset0))
{
	vR[i] = 1797;
}
else
{
	vR[i] = 1796;
}

}

}
else
{
	if(input[i]>(1793+offset0))
{
	if(input[i]==(1795+offset0))
{
	vR[i] = 1795;
}
else
{
	vR[i] = 1794;
}

}
else
{
	if(input[i]==(1793+offset0))
{
	vR[i] = 1793;
}
else
{
	vR[i] = 1792;
}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(1663+offset0))
{
	if(input[i]>(1727+offset0))
{
	if(input[i]>(1759+offset0))
{
	if(input[i]>(1775+offset0))
{
	if(input[i]>(1783+offset0))
{
	if(input[i]>(1787+offset0))
{
	if(input[i]>(1789+offset0))
{
	if(input[i]==(1791+offset0))
{
	vR[i] = 1791;
}
else
{
	vR[i] = 1790;
}

}
else
{
	if(input[i]==(1789+offset0))
{
	vR[i] = 1789;
}
else
{
	vR[i] = 1788;
}

}

}
else
{
	if(input[i]>(1785+offset0))
{
	if(input[i]==(1787+offset0))
{
	vR[i] = 1787;
}
else
{
	vR[i] = 1786;
}

}
else
{
	if(input[i]==(1785+offset0))
{
	vR[i] = 1785;
}
else
{
	vR[i] = 1784;
}

}

}

}
else
{
	if(input[i]>(1779+offset0))
{
	if(input[i]>(1781+offset0))
{
	if(input[i]==(1783+offset0))
{
	vR[i] = 1783;
}
else
{
	vR[i] = 1782;
}

}
else
{
	if(input[i]==(1781+offset0))
{
	vR[i] = 1781;
}
else
{
	vR[i] = 1780;
}

}

}
else
{
	if(input[i]>(1777+offset0))
{
	if(input[i]==(1779+offset0))
{
	vR[i] = 1779;
}
else
{
	vR[i] = 1778;
}

}
else
{
	if(input[i]==(1777+offset0))
{
	vR[i] = 1777;
}
else
{
	vR[i] = 1776;
}

}

}

}

}
else
{
	if(input[i]>(1767+offset0))
{
	if(input[i]>(1771+offset0))
{
	if(input[i]>(1773+offset0))
{
	if(input[i]==(1775+offset0))
{
	vR[i] = 1775;
}
else
{
	vR[i] = 1774;
}

}
else
{
	if(input[i]==(1773+offset0))
{
	vR[i] = 1773;
}
else
{
	vR[i] = 1772;
}

}

}
else
{
	if(input[i]>(1769+offset0))
{
	if(input[i]==(1771+offset0))
{
	vR[i] = 1771;
}
else
{
	vR[i] = 1770;
}

}
else
{
	if(input[i]==(1769+offset0))
{
	vR[i] = 1769;
}
else
{
	vR[i] = 1768;
}

}

}

}
else
{
	if(input[i]>(1763+offset0))
{
	if(input[i]>(1765+offset0))
{
	if(input[i]==(1767+offset0))
{
	vR[i] = 1767;
}
else
{
	vR[i] = 1766;
}

}
else
{
	if(input[i]==(1765+offset0))
{
	vR[i] = 1765;
}
else
{
	vR[i] = 1764;
}

}

}
else
{
	if(input[i]>(1761+offset0))
{
	if(input[i]==(1763+offset0))
{
	vR[i] = 1763;
}
else
{
	vR[i] = 1762;
}

}
else
{
	if(input[i]==(1761+offset0))
{
	vR[i] = 1761;
}
else
{
	vR[i] = 1760;
}

}

}

}

}

}
else
{
	if(input[i]>(1743+offset0))
{
	if(input[i]>(1751+offset0))
{
	if(input[i]>(1755+offset0))
{
	if(input[i]>(1757+offset0))
{
	if(input[i]==(1759+offset0))
{
	vR[i] = 1759;
}
else
{
	vR[i] = 1758;
}

}
else
{
	if(input[i]==(1757+offset0))
{
	vR[i] = 1757;
}
else
{
	vR[i] = 1756;
}

}

}
else
{
	if(input[i]>(1753+offset0))
{
	if(input[i]==(1755+offset0))
{
	vR[i] = 1755;
}
else
{
	vR[i] = 1754;
}

}
else
{
	if(input[i]==(1753+offset0))
{
	vR[i] = 1753;
}
else
{
	vR[i] = 1752;
}

}

}

}
else
{
	if(input[i]>(1747+offset0))
{
	if(input[i]>(1749+offset0))
{
	if(input[i]==(1751+offset0))
{
	vR[i] = 1751;
}
else
{
	vR[i] = 1750;
}

}
else
{
	if(input[i]==(1749+offset0))
{
	vR[i] = 1749;
}
else
{
	vR[i] = 1748;
}

}

}
else
{
	if(input[i]>(1745+offset0))
{
	if(input[i]==(1747+offset0))
{
	vR[i] = 1747;
}
else
{
	vR[i] = 1746;
}

}
else
{
	if(input[i]==(1745+offset0))
{
	vR[i] = 1745;
}
else
{
	vR[i] = 1744;
}

}

}

}

}
else
{
	if(input[i]>(1735+offset0))
{
	if(input[i]>(1739+offset0))
{
	if(input[i]>(1741+offset0))
{
	if(input[i]==(1743+offset0))
{
	vR[i] = 1743;
}
else
{
	vR[i] = 1742;
}

}
else
{
	if(input[i]==(1741+offset0))
{
	vR[i] = 1741;
}
else
{
	vR[i] = 1740;
}

}

}
else
{
	if(input[i]>(1737+offset0))
{
	if(input[i]==(1739+offset0))
{
	vR[i] = 1739;
}
else
{
	vR[i] = 1738;
}

}
else
{
	if(input[i]==(1737+offset0))
{
	vR[i] = 1737;
}
else
{
	vR[i] = 1736;
}

}

}

}
else
{
	if(input[i]>(1731+offset0))
{
	if(input[i]>(1733+offset0))
{
	if(input[i]==(1735+offset0))
{
	vR[i] = 1735;
}
else
{
	vR[i] = 1734;
}

}
else
{
	if(input[i]==(1733+offset0))
{
	vR[i] = 1733;
}
else
{
	vR[i] = 1732;
}

}

}
else
{
	if(input[i]>(1729+offset0))
{
	if(input[i]==(1731+offset0))
{
	vR[i] = 1731;
}
else
{
	vR[i] = 1730;
}

}
else
{
	if(input[i]==(1729+offset0))
{
	vR[i] = 1729;
}
else
{
	vR[i] = 1728;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1695+offset0))
{
	if(input[i]>(1711+offset0))
{
	if(input[i]>(1719+offset0))
{
	if(input[i]>(1723+offset0))
{
	if(input[i]>(1725+offset0))
{
	if(input[i]==(1727+offset0))
{
	vR[i] = 1727;
}
else
{
	vR[i] = 1726;
}

}
else
{
	if(input[i]==(1725+offset0))
{
	vR[i] = 1725;
}
else
{
	vR[i] = 1724;
}

}

}
else
{
	if(input[i]>(1721+offset0))
{
	if(input[i]==(1723+offset0))
{
	vR[i] = 1723;
}
else
{
	vR[i] = 1722;
}

}
else
{
	if(input[i]==(1721+offset0))
{
	vR[i] = 1721;
}
else
{
	vR[i] = 1720;
}

}

}

}
else
{
	if(input[i]>(1715+offset0))
{
	if(input[i]>(1717+offset0))
{
	if(input[i]==(1719+offset0))
{
	vR[i] = 1719;
}
else
{
	vR[i] = 1718;
}

}
else
{
	if(input[i]==(1717+offset0))
{
	vR[i] = 1717;
}
else
{
	vR[i] = 1716;
}

}

}
else
{
	if(input[i]>(1713+offset0))
{
	if(input[i]==(1715+offset0))
{
	vR[i] = 1715;
}
else
{
	vR[i] = 1714;
}

}
else
{
	if(input[i]==(1713+offset0))
{
	vR[i] = 1713;
}
else
{
	vR[i] = 1712;
}

}

}

}

}
else
{
	if(input[i]>(1703+offset0))
{
	if(input[i]>(1707+offset0))
{
	if(input[i]>(1709+offset0))
{
	if(input[i]==(1711+offset0))
{
	vR[i] = 1711;
}
else
{
	vR[i] = 1710;
}

}
else
{
	if(input[i]==(1709+offset0))
{
	vR[i] = 1709;
}
else
{
	vR[i] = 1708;
}

}

}
else
{
	if(input[i]>(1705+offset0))
{
	if(input[i]==(1707+offset0))
{
	vR[i] = 1707;
}
else
{
	vR[i] = 1706;
}

}
else
{
	if(input[i]==(1705+offset0))
{
	vR[i] = 1705;
}
else
{
	vR[i] = 1704;
}

}

}

}
else
{
	if(input[i]>(1699+offset0))
{
	if(input[i]>(1701+offset0))
{
	if(input[i]==(1703+offset0))
{
	vR[i] = 1703;
}
else
{
	vR[i] = 1702;
}

}
else
{
	if(input[i]==(1701+offset0))
{
	vR[i] = 1701;
}
else
{
	vR[i] = 1700;
}

}

}
else
{
	if(input[i]>(1697+offset0))
{
	if(input[i]==(1699+offset0))
{
	vR[i] = 1699;
}
else
{
	vR[i] = 1698;
}

}
else
{
	if(input[i]==(1697+offset0))
{
	vR[i] = 1697;
}
else
{
	vR[i] = 1696;
}

}

}

}

}

}
else
{
	if(input[i]>(1679+offset0))
{
	if(input[i]>(1687+offset0))
{
	if(input[i]>(1691+offset0))
{
	if(input[i]>(1693+offset0))
{
	if(input[i]==(1695+offset0))
{
	vR[i] = 1695;
}
else
{
	vR[i] = 1694;
}

}
else
{
	if(input[i]==(1693+offset0))
{
	vR[i] = 1693;
}
else
{
	vR[i] = 1692;
}

}

}
else
{
	if(input[i]>(1689+offset0))
{
	if(input[i]==(1691+offset0))
{
	vR[i] = 1691;
}
else
{
	vR[i] = 1690;
}

}
else
{
	if(input[i]==(1689+offset0))
{
	vR[i] = 1689;
}
else
{
	vR[i] = 1688;
}

}

}

}
else
{
	if(input[i]>(1683+offset0))
{
	if(input[i]>(1685+offset0))
{
	if(input[i]==(1687+offset0))
{
	vR[i] = 1687;
}
else
{
	vR[i] = 1686;
}

}
else
{
	if(input[i]==(1685+offset0))
{
	vR[i] = 1685;
}
else
{
	vR[i] = 1684;
}

}

}
else
{
	if(input[i]>(1681+offset0))
{
	if(input[i]==(1683+offset0))
{
	vR[i] = 1683;
}
else
{
	vR[i] = 1682;
}

}
else
{
	if(input[i]==(1681+offset0))
{
	vR[i] = 1681;
}
else
{
	vR[i] = 1680;
}

}

}

}

}
else
{
	if(input[i]>(1671+offset0))
{
	if(input[i]>(1675+offset0))
{
	if(input[i]>(1677+offset0))
{
	if(input[i]==(1679+offset0))
{
	vR[i] = 1679;
}
else
{
	vR[i] = 1678;
}

}
else
{
	if(input[i]==(1677+offset0))
{
	vR[i] = 1677;
}
else
{
	vR[i] = 1676;
}

}

}
else
{
	if(input[i]>(1673+offset0))
{
	if(input[i]==(1675+offset0))
{
	vR[i] = 1675;
}
else
{
	vR[i] = 1674;
}

}
else
{
	if(input[i]==(1673+offset0))
{
	vR[i] = 1673;
}
else
{
	vR[i] = 1672;
}

}

}

}
else
{
	if(input[i]>(1667+offset0))
{
	if(input[i]>(1669+offset0))
{
	if(input[i]==(1671+offset0))
{
	vR[i] = 1671;
}
else
{
	vR[i] = 1670;
}

}
else
{
	if(input[i]==(1669+offset0))
{
	vR[i] = 1669;
}
else
{
	vR[i] = 1668;
}

}

}
else
{
	if(input[i]>(1665+offset0))
{
	if(input[i]==(1667+offset0))
{
	vR[i] = 1667;
}
else
{
	vR[i] = 1666;
}

}
else
{
	if(input[i]==(1665+offset0))
{
	vR[i] = 1665;
}
else
{
	vR[i] = 1664;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(1599+offset0))
{
	if(input[i]>(1631+offset0))
{
	if(input[i]>(1647+offset0))
{
	if(input[i]>(1655+offset0))
{
	if(input[i]>(1659+offset0))
{
	if(input[i]>(1661+offset0))
{
	if(input[i]==(1663+offset0))
{
	vR[i] = 1663;
}
else
{
	vR[i] = 1662;
}

}
else
{
	if(input[i]==(1661+offset0))
{
	vR[i] = 1661;
}
else
{
	vR[i] = 1660;
}

}

}
else
{
	if(input[i]>(1657+offset0))
{
	if(input[i]==(1659+offset0))
{
	vR[i] = 1659;
}
else
{
	vR[i] = 1658;
}

}
else
{
	if(input[i]==(1657+offset0))
{
	vR[i] = 1657;
}
else
{
	vR[i] = 1656;
}

}

}

}
else
{
	if(input[i]>(1651+offset0))
{
	if(input[i]>(1653+offset0))
{
	if(input[i]==(1655+offset0))
{
	vR[i] = 1655;
}
else
{
	vR[i] = 1654;
}

}
else
{
	if(input[i]==(1653+offset0))
{
	vR[i] = 1653;
}
else
{
	vR[i] = 1652;
}

}

}
else
{
	if(input[i]>(1649+offset0))
{
	if(input[i]==(1651+offset0))
{
	vR[i] = 1651;
}
else
{
	vR[i] = 1650;
}

}
else
{
	if(input[i]==(1649+offset0))
{
	vR[i] = 1649;
}
else
{
	vR[i] = 1648;
}

}

}

}

}
else
{
	if(input[i]>(1639+offset0))
{
	if(input[i]>(1643+offset0))
{
	if(input[i]>(1645+offset0))
{
	if(input[i]==(1647+offset0))
{
	vR[i] = 1647;
}
else
{
	vR[i] = 1646;
}

}
else
{
	if(input[i]==(1645+offset0))
{
	vR[i] = 1645;
}
else
{
	vR[i] = 1644;
}

}

}
else
{
	if(input[i]>(1641+offset0))
{
	if(input[i]==(1643+offset0))
{
	vR[i] = 1643;
}
else
{
	vR[i] = 1642;
}

}
else
{
	if(input[i]==(1641+offset0))
{
	vR[i] = 1641;
}
else
{
	vR[i] = 1640;
}

}

}

}
else
{
	if(input[i]>(1635+offset0))
{
	if(input[i]>(1637+offset0))
{
	if(input[i]==(1639+offset0))
{
	vR[i] = 1639;
}
else
{
	vR[i] = 1638;
}

}
else
{
	if(input[i]==(1637+offset0))
{
	vR[i] = 1637;
}
else
{
	vR[i] = 1636;
}

}

}
else
{
	if(input[i]>(1633+offset0))
{
	if(input[i]==(1635+offset0))
{
	vR[i] = 1635;
}
else
{
	vR[i] = 1634;
}

}
else
{
	if(input[i]==(1633+offset0))
{
	vR[i] = 1633;
}
else
{
	vR[i] = 1632;
}

}

}

}

}

}
else
{
	if(input[i]>(1615+offset0))
{
	if(input[i]>(1623+offset0))
{
	if(input[i]>(1627+offset0))
{
	if(input[i]>(1629+offset0))
{
	if(input[i]==(1631+offset0))
{
	vR[i] = 1631;
}
else
{
	vR[i] = 1630;
}

}
else
{
	if(input[i]==(1629+offset0))
{
	vR[i] = 1629;
}
else
{
	vR[i] = 1628;
}

}

}
else
{
	if(input[i]>(1625+offset0))
{
	if(input[i]==(1627+offset0))
{
	vR[i] = 1627;
}
else
{
	vR[i] = 1626;
}

}
else
{
	if(input[i]==(1625+offset0))
{
	vR[i] = 1625;
}
else
{
	vR[i] = 1624;
}

}

}

}
else
{
	if(input[i]>(1619+offset0))
{
	if(input[i]>(1621+offset0))
{
	if(input[i]==(1623+offset0))
{
	vR[i] = 1623;
}
else
{
	vR[i] = 1622;
}

}
else
{
	if(input[i]==(1621+offset0))
{
	vR[i] = 1621;
}
else
{
	vR[i] = 1620;
}

}

}
else
{
	if(input[i]>(1617+offset0))
{
	if(input[i]==(1619+offset0))
{
	vR[i] = 1619;
}
else
{
	vR[i] = 1618;
}

}
else
{
	if(input[i]==(1617+offset0))
{
	vR[i] = 1617;
}
else
{
	vR[i] = 1616;
}

}

}

}

}
else
{
	if(input[i]>(1607+offset0))
{
	if(input[i]>(1611+offset0))
{
	if(input[i]>(1613+offset0))
{
	if(input[i]==(1615+offset0))
{
	vR[i] = 1615;
}
else
{
	vR[i] = 1614;
}

}
else
{
	if(input[i]==(1613+offset0))
{
	vR[i] = 1613;
}
else
{
	vR[i] = 1612;
}

}

}
else
{
	if(input[i]>(1609+offset0))
{
	if(input[i]==(1611+offset0))
{
	vR[i] = 1611;
}
else
{
	vR[i] = 1610;
}

}
else
{
	if(input[i]==(1609+offset0))
{
	vR[i] = 1609;
}
else
{
	vR[i] = 1608;
}

}

}

}
else
{
	if(input[i]>(1603+offset0))
{
	if(input[i]>(1605+offset0))
{
	if(input[i]==(1607+offset0))
{
	vR[i] = 1607;
}
else
{
	vR[i] = 1606;
}

}
else
{
	if(input[i]==(1605+offset0))
{
	vR[i] = 1605;
}
else
{
	vR[i] = 1604;
}

}

}
else
{
	if(input[i]>(1601+offset0))
{
	if(input[i]==(1603+offset0))
{
	vR[i] = 1603;
}
else
{
	vR[i] = 1602;
}

}
else
{
	if(input[i]==(1601+offset0))
{
	vR[i] = 1601;
}
else
{
	vR[i] = 1600;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1567+offset0))
{
	if(input[i]>(1583+offset0))
{
	if(input[i]>(1591+offset0))
{
	if(input[i]>(1595+offset0))
{
	if(input[i]>(1597+offset0))
{
	if(input[i]==(1599+offset0))
{
	vR[i] = 1599;
}
else
{
	vR[i] = 1598;
}

}
else
{
	if(input[i]==(1597+offset0))
{
	vR[i] = 1597;
}
else
{
	vR[i] = 1596;
}

}

}
else
{
	if(input[i]>(1593+offset0))
{
	if(input[i]==(1595+offset0))
{
	vR[i] = 1595;
}
else
{
	vR[i] = 1594;
}

}
else
{
	if(input[i]==(1593+offset0))
{
	vR[i] = 1593;
}
else
{
	vR[i] = 1592;
}

}

}

}
else
{
	if(input[i]>(1587+offset0))
{
	if(input[i]>(1589+offset0))
{
	if(input[i]==(1591+offset0))
{
	vR[i] = 1591;
}
else
{
	vR[i] = 1590;
}

}
else
{
	if(input[i]==(1589+offset0))
{
	vR[i] = 1589;
}
else
{
	vR[i] = 1588;
}

}

}
else
{
	if(input[i]>(1585+offset0))
{
	if(input[i]==(1587+offset0))
{
	vR[i] = 1587;
}
else
{
	vR[i] = 1586;
}

}
else
{
	if(input[i]==(1585+offset0))
{
	vR[i] = 1585;
}
else
{
	vR[i] = 1584;
}

}

}

}

}
else
{
	if(input[i]>(1575+offset0))
{
	if(input[i]>(1579+offset0))
{
	if(input[i]>(1581+offset0))
{
	if(input[i]==(1583+offset0))
{
	vR[i] = 1583;
}
else
{
	vR[i] = 1582;
}

}
else
{
	if(input[i]==(1581+offset0))
{
	vR[i] = 1581;
}
else
{
	vR[i] = 1580;
}

}

}
else
{
	if(input[i]>(1577+offset0))
{
	if(input[i]==(1579+offset0))
{
	vR[i] = 1579;
}
else
{
	vR[i] = 1578;
}

}
else
{
	if(input[i]==(1577+offset0))
{
	vR[i] = 1577;
}
else
{
	vR[i] = 1576;
}

}

}

}
else
{
	if(input[i]>(1571+offset0))
{
	if(input[i]>(1573+offset0))
{
	if(input[i]==(1575+offset0))
{
	vR[i] = 1575;
}
else
{
	vR[i] = 1574;
}

}
else
{
	if(input[i]==(1573+offset0))
{
	vR[i] = 1573;
}
else
{
	vR[i] = 1572;
}

}

}
else
{
	if(input[i]>(1569+offset0))
{
	if(input[i]==(1571+offset0))
{
	vR[i] = 1571;
}
else
{
	vR[i] = 1570;
}

}
else
{
	if(input[i]==(1569+offset0))
{
	vR[i] = 1569;
}
else
{
	vR[i] = 1568;
}

}

}

}

}

}
else
{
	if(input[i]>(1551+offset0))
{
	if(input[i]>(1559+offset0))
{
	if(input[i]>(1563+offset0))
{
	if(input[i]>(1565+offset0))
{
	if(input[i]==(1567+offset0))
{
	vR[i] = 1567;
}
else
{
	vR[i] = 1566;
}

}
else
{
	if(input[i]==(1565+offset0))
{
	vR[i] = 1565;
}
else
{
	vR[i] = 1564;
}

}

}
else
{
	if(input[i]>(1561+offset0))
{
	if(input[i]==(1563+offset0))
{
	vR[i] = 1563;
}
else
{
	vR[i] = 1562;
}

}
else
{
	if(input[i]==(1561+offset0))
{
	vR[i] = 1561;
}
else
{
	vR[i] = 1560;
}

}

}

}
else
{
	if(input[i]>(1555+offset0))
{
	if(input[i]>(1557+offset0))
{
	if(input[i]==(1559+offset0))
{
	vR[i] = 1559;
}
else
{
	vR[i] = 1558;
}

}
else
{
	if(input[i]==(1557+offset0))
{
	vR[i] = 1557;
}
else
{
	vR[i] = 1556;
}

}

}
else
{
	if(input[i]>(1553+offset0))
{
	if(input[i]==(1555+offset0))
{
	vR[i] = 1555;
}
else
{
	vR[i] = 1554;
}

}
else
{
	if(input[i]==(1553+offset0))
{
	vR[i] = 1553;
}
else
{
	vR[i] = 1552;
}

}

}

}

}
else
{
	if(input[i]>(1543+offset0))
{
	if(input[i]>(1547+offset0))
{
	if(input[i]>(1549+offset0))
{
	if(input[i]==(1551+offset0))
{
	vR[i] = 1551;
}
else
{
	vR[i] = 1550;
}

}
else
{
	if(input[i]==(1549+offset0))
{
	vR[i] = 1549;
}
else
{
	vR[i] = 1548;
}

}

}
else
{
	if(input[i]>(1545+offset0))
{
	if(input[i]==(1547+offset0))
{
	vR[i] = 1547;
}
else
{
	vR[i] = 1546;
}

}
else
{
	if(input[i]==(1545+offset0))
{
	vR[i] = 1545;
}
else
{
	vR[i] = 1544;
}

}

}

}
else
{
	if(input[i]>(1539+offset0))
{
	if(input[i]>(1541+offset0))
{
	if(input[i]==(1543+offset0))
{
	vR[i] = 1543;
}
else
{
	vR[i] = 1542;
}

}
else
{
	if(input[i]==(1541+offset0))
{
	vR[i] = 1541;
}
else
{
	vR[i] = 1540;
}

}

}
else
{
	if(input[i]>(1537+offset0))
{
	if(input[i]==(1539+offset0))
{
	vR[i] = 1539;
}
else
{
	vR[i] = 1538;
}

}
else
{
	if(input[i]==(1537+offset0))
{
	vR[i] = 1537;
}
else
{
	vR[i] = 1536;
}

}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(1279+offset0))
{
	if(input[i]>(1407+offset0))
{
	if(input[i]>(1471+offset0))
{
	if(input[i]>(1503+offset0))
{
	if(input[i]>(1519+offset0))
{
	if(input[i]>(1527+offset0))
{
	if(input[i]>(1531+offset0))
{
	if(input[i]>(1533+offset0))
{
	if(input[i]==(1535+offset0))
{
	vR[i] = 1535;
}
else
{
	vR[i] = 1534;
}

}
else
{
	if(input[i]==(1533+offset0))
{
	vR[i] = 1533;
}
else
{
	vR[i] = 1532;
}

}

}
else
{
	if(input[i]>(1529+offset0))
{
	if(input[i]==(1531+offset0))
{
	vR[i] = 1531;
}
else
{
	vR[i] = 1530;
}

}
else
{
	if(input[i]==(1529+offset0))
{
	vR[i] = 1529;
}
else
{
	vR[i] = 1528;
}

}

}

}
else
{
	if(input[i]>(1523+offset0))
{
	if(input[i]>(1525+offset0))
{
	if(input[i]==(1527+offset0))
{
	vR[i] = 1527;
}
else
{
	vR[i] = 1526;
}

}
else
{
	if(input[i]==(1525+offset0))
{
	vR[i] = 1525;
}
else
{
	vR[i] = 1524;
}

}

}
else
{
	if(input[i]>(1521+offset0))
{
	if(input[i]==(1523+offset0))
{
	vR[i] = 1523;
}
else
{
	vR[i] = 1522;
}

}
else
{
	if(input[i]==(1521+offset0))
{
	vR[i] = 1521;
}
else
{
	vR[i] = 1520;
}

}

}

}

}
else
{
	if(input[i]>(1511+offset0))
{
	if(input[i]>(1515+offset0))
{
	if(input[i]>(1517+offset0))
{
	if(input[i]==(1519+offset0))
{
	vR[i] = 1519;
}
else
{
	vR[i] = 1518;
}

}
else
{
	if(input[i]==(1517+offset0))
{
	vR[i] = 1517;
}
else
{
	vR[i] = 1516;
}

}

}
else
{
	if(input[i]>(1513+offset0))
{
	if(input[i]==(1515+offset0))
{
	vR[i] = 1515;
}
else
{
	vR[i] = 1514;
}

}
else
{
	if(input[i]==(1513+offset0))
{
	vR[i] = 1513;
}
else
{
	vR[i] = 1512;
}

}

}

}
else
{
	if(input[i]>(1507+offset0))
{
	if(input[i]>(1509+offset0))
{
	if(input[i]==(1511+offset0))
{
	vR[i] = 1511;
}
else
{
	vR[i] = 1510;
}

}
else
{
	if(input[i]==(1509+offset0))
{
	vR[i] = 1509;
}
else
{
	vR[i] = 1508;
}

}

}
else
{
	if(input[i]>(1505+offset0))
{
	if(input[i]==(1507+offset0))
{
	vR[i] = 1507;
}
else
{
	vR[i] = 1506;
}

}
else
{
	if(input[i]==(1505+offset0))
{
	vR[i] = 1505;
}
else
{
	vR[i] = 1504;
}

}

}

}

}

}
else
{
	if(input[i]>(1487+offset0))
{
	if(input[i]>(1495+offset0))
{
	if(input[i]>(1499+offset0))
{
	if(input[i]>(1501+offset0))
{
	if(input[i]==(1503+offset0))
{
	vR[i] = 1503;
}
else
{
	vR[i] = 1502;
}

}
else
{
	if(input[i]==(1501+offset0))
{
	vR[i] = 1501;
}
else
{
	vR[i] = 1500;
}

}

}
else
{
	if(input[i]>(1497+offset0))
{
	if(input[i]==(1499+offset0))
{
	vR[i] = 1499;
}
else
{
	vR[i] = 1498;
}

}
else
{
	if(input[i]==(1497+offset0))
{
	vR[i] = 1497;
}
else
{
	vR[i] = 1496;
}

}

}

}
else
{
	if(input[i]>(1491+offset0))
{
	if(input[i]>(1493+offset0))
{
	if(input[i]==(1495+offset0))
{
	vR[i] = 1495;
}
else
{
	vR[i] = 1494;
}

}
else
{
	if(input[i]==(1493+offset0))
{
	vR[i] = 1493;
}
else
{
	vR[i] = 1492;
}

}

}
else
{
	if(input[i]>(1489+offset0))
{
	if(input[i]==(1491+offset0))
{
	vR[i] = 1491;
}
else
{
	vR[i] = 1490;
}

}
else
{
	if(input[i]==(1489+offset0))
{
	vR[i] = 1489;
}
else
{
	vR[i] = 1488;
}

}

}

}

}
else
{
	if(input[i]>(1479+offset0))
{
	if(input[i]>(1483+offset0))
{
	if(input[i]>(1485+offset0))
{
	if(input[i]==(1487+offset0))
{
	vR[i] = 1487;
}
else
{
	vR[i] = 1486;
}

}
else
{
	if(input[i]==(1485+offset0))
{
	vR[i] = 1485;
}
else
{
	vR[i] = 1484;
}

}

}
else
{
	if(input[i]>(1481+offset0))
{
	if(input[i]==(1483+offset0))
{
	vR[i] = 1483;
}
else
{
	vR[i] = 1482;
}

}
else
{
	if(input[i]==(1481+offset0))
{
	vR[i] = 1481;
}
else
{
	vR[i] = 1480;
}

}

}

}
else
{
	if(input[i]>(1475+offset0))
{
	if(input[i]>(1477+offset0))
{
	if(input[i]==(1479+offset0))
{
	vR[i] = 1479;
}
else
{
	vR[i] = 1478;
}

}
else
{
	if(input[i]==(1477+offset0))
{
	vR[i] = 1477;
}
else
{
	vR[i] = 1476;
}

}

}
else
{
	if(input[i]>(1473+offset0))
{
	if(input[i]==(1475+offset0))
{
	vR[i] = 1475;
}
else
{
	vR[i] = 1474;
}

}
else
{
	if(input[i]==(1473+offset0))
{
	vR[i] = 1473;
}
else
{
	vR[i] = 1472;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1439+offset0))
{
	if(input[i]>(1455+offset0))
{
	if(input[i]>(1463+offset0))
{
	if(input[i]>(1467+offset0))
{
	if(input[i]>(1469+offset0))
{
	if(input[i]==(1471+offset0))
{
	vR[i] = 1471;
}
else
{
	vR[i] = 1470;
}

}
else
{
	if(input[i]==(1469+offset0))
{
	vR[i] = 1469;
}
else
{
	vR[i] = 1468;
}

}

}
else
{
	if(input[i]>(1465+offset0))
{
	if(input[i]==(1467+offset0))
{
	vR[i] = 1467;
}
else
{
	vR[i] = 1466;
}

}
else
{
	if(input[i]==(1465+offset0))
{
	vR[i] = 1465;
}
else
{
	vR[i] = 1464;
}

}

}

}
else
{
	if(input[i]>(1459+offset0))
{
	if(input[i]>(1461+offset0))
{
	if(input[i]==(1463+offset0))
{
	vR[i] = 1463;
}
else
{
	vR[i] = 1462;
}

}
else
{
	if(input[i]==(1461+offset0))
{
	vR[i] = 1461;
}
else
{
	vR[i] = 1460;
}

}

}
else
{
	if(input[i]>(1457+offset0))
{
	if(input[i]==(1459+offset0))
{
	vR[i] = 1459;
}
else
{
	vR[i] = 1458;
}

}
else
{
	if(input[i]==(1457+offset0))
{
	vR[i] = 1457;
}
else
{
	vR[i] = 1456;
}

}

}

}

}
else
{
	if(input[i]>(1447+offset0))
{
	if(input[i]>(1451+offset0))
{
	if(input[i]>(1453+offset0))
{
	if(input[i]==(1455+offset0))
{
	vR[i] = 1455;
}
else
{
	vR[i] = 1454;
}

}
else
{
	if(input[i]==(1453+offset0))
{
	vR[i] = 1453;
}
else
{
	vR[i] = 1452;
}

}

}
else
{
	if(input[i]>(1449+offset0))
{
	if(input[i]==(1451+offset0))
{
	vR[i] = 1451;
}
else
{
	vR[i] = 1450;
}

}
else
{
	if(input[i]==(1449+offset0))
{
	vR[i] = 1449;
}
else
{
	vR[i] = 1448;
}

}

}

}
else
{
	if(input[i]>(1443+offset0))
{
	if(input[i]>(1445+offset0))
{
	if(input[i]==(1447+offset0))
{
	vR[i] = 1447;
}
else
{
	vR[i] = 1446;
}

}
else
{
	if(input[i]==(1445+offset0))
{
	vR[i] = 1445;
}
else
{
	vR[i] = 1444;
}

}

}
else
{
	if(input[i]>(1441+offset0))
{
	if(input[i]==(1443+offset0))
{
	vR[i] = 1443;
}
else
{
	vR[i] = 1442;
}

}
else
{
	if(input[i]==(1441+offset0))
{
	vR[i] = 1441;
}
else
{
	vR[i] = 1440;
}

}

}

}

}

}
else
{
	if(input[i]>(1423+offset0))
{
	if(input[i]>(1431+offset0))
{
	if(input[i]>(1435+offset0))
{
	if(input[i]>(1437+offset0))
{
	if(input[i]==(1439+offset0))
{
	vR[i] = 1439;
}
else
{
	vR[i] = 1438;
}

}
else
{
	if(input[i]==(1437+offset0))
{
	vR[i] = 1437;
}
else
{
	vR[i] = 1436;
}

}

}
else
{
	if(input[i]>(1433+offset0))
{
	if(input[i]==(1435+offset0))
{
	vR[i] = 1435;
}
else
{
	vR[i] = 1434;
}

}
else
{
	if(input[i]==(1433+offset0))
{
	vR[i] = 1433;
}
else
{
	vR[i] = 1432;
}

}

}

}
else
{
	if(input[i]>(1427+offset0))
{
	if(input[i]>(1429+offset0))
{
	if(input[i]==(1431+offset0))
{
	vR[i] = 1431;
}
else
{
	vR[i] = 1430;
}

}
else
{
	if(input[i]==(1429+offset0))
{
	vR[i] = 1429;
}
else
{
	vR[i] = 1428;
}

}

}
else
{
	if(input[i]>(1425+offset0))
{
	if(input[i]==(1427+offset0))
{
	vR[i] = 1427;
}
else
{
	vR[i] = 1426;
}

}
else
{
	if(input[i]==(1425+offset0))
{
	vR[i] = 1425;
}
else
{
	vR[i] = 1424;
}

}

}

}

}
else
{
	if(input[i]>(1415+offset0))
{
	if(input[i]>(1419+offset0))
{
	if(input[i]>(1421+offset0))
{
	if(input[i]==(1423+offset0))
{
	vR[i] = 1423;
}
else
{
	vR[i] = 1422;
}

}
else
{
	if(input[i]==(1421+offset0))
{
	vR[i] = 1421;
}
else
{
	vR[i] = 1420;
}

}

}
else
{
	if(input[i]>(1417+offset0))
{
	if(input[i]==(1419+offset0))
{
	vR[i] = 1419;
}
else
{
	vR[i] = 1418;
}

}
else
{
	if(input[i]==(1417+offset0))
{
	vR[i] = 1417;
}
else
{
	vR[i] = 1416;
}

}

}

}
else
{
	if(input[i]>(1411+offset0))
{
	if(input[i]>(1413+offset0))
{
	if(input[i]==(1415+offset0))
{
	vR[i] = 1415;
}
else
{
	vR[i] = 1414;
}

}
else
{
	if(input[i]==(1413+offset0))
{
	vR[i] = 1413;
}
else
{
	vR[i] = 1412;
}

}

}
else
{
	if(input[i]>(1409+offset0))
{
	if(input[i]==(1411+offset0))
{
	vR[i] = 1411;
}
else
{
	vR[i] = 1410;
}

}
else
{
	if(input[i]==(1409+offset0))
{
	vR[i] = 1409;
}
else
{
	vR[i] = 1408;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(1343+offset0))
{
	if(input[i]>(1375+offset0))
{
	if(input[i]>(1391+offset0))
{
	if(input[i]>(1399+offset0))
{
	if(input[i]>(1403+offset0))
{
	if(input[i]>(1405+offset0))
{
	if(input[i]==(1407+offset0))
{
	vR[i] = 1407;
}
else
{
	vR[i] = 1406;
}

}
else
{
	if(input[i]==(1405+offset0))
{
	vR[i] = 1405;
}
else
{
	vR[i] = 1404;
}

}

}
else
{
	if(input[i]>(1401+offset0))
{
	if(input[i]==(1403+offset0))
{
	vR[i] = 1403;
}
else
{
	vR[i] = 1402;
}

}
else
{
	if(input[i]==(1401+offset0))
{
	vR[i] = 1401;
}
else
{
	vR[i] = 1400;
}

}

}

}
else
{
	if(input[i]>(1395+offset0))
{
	if(input[i]>(1397+offset0))
{
	if(input[i]==(1399+offset0))
{
	vR[i] = 1399;
}
else
{
	vR[i] = 1398;
}

}
else
{
	if(input[i]==(1397+offset0))
{
	vR[i] = 1397;
}
else
{
	vR[i] = 1396;
}

}

}
else
{
	if(input[i]>(1393+offset0))
{
	if(input[i]==(1395+offset0))
{
	vR[i] = 1395;
}
else
{
	vR[i] = 1394;
}

}
else
{
	if(input[i]==(1393+offset0))
{
	vR[i] = 1393;
}
else
{
	vR[i] = 1392;
}

}

}

}

}
else
{
	if(input[i]>(1383+offset0))
{
	if(input[i]>(1387+offset0))
{
	if(input[i]>(1389+offset0))
{
	if(input[i]==(1391+offset0))
{
	vR[i] = 1391;
}
else
{
	vR[i] = 1390;
}

}
else
{
	if(input[i]==(1389+offset0))
{
	vR[i] = 1389;
}
else
{
	vR[i] = 1388;
}

}

}
else
{
	if(input[i]>(1385+offset0))
{
	if(input[i]==(1387+offset0))
{
	vR[i] = 1387;
}
else
{
	vR[i] = 1386;
}

}
else
{
	if(input[i]==(1385+offset0))
{
	vR[i] = 1385;
}
else
{
	vR[i] = 1384;
}

}

}

}
else
{
	if(input[i]>(1379+offset0))
{
	if(input[i]>(1381+offset0))
{
	if(input[i]==(1383+offset0))
{
	vR[i] = 1383;
}
else
{
	vR[i] = 1382;
}

}
else
{
	if(input[i]==(1381+offset0))
{
	vR[i] = 1381;
}
else
{
	vR[i] = 1380;
}

}

}
else
{
	if(input[i]>(1377+offset0))
{
	if(input[i]==(1379+offset0))
{
	vR[i] = 1379;
}
else
{
	vR[i] = 1378;
}

}
else
{
	if(input[i]==(1377+offset0))
{
	vR[i] = 1377;
}
else
{
	vR[i] = 1376;
}

}

}

}

}

}
else
{
	if(input[i]>(1359+offset0))
{
	if(input[i]>(1367+offset0))
{
	if(input[i]>(1371+offset0))
{
	if(input[i]>(1373+offset0))
{
	if(input[i]==(1375+offset0))
{
	vR[i] = 1375;
}
else
{
	vR[i] = 1374;
}

}
else
{
	if(input[i]==(1373+offset0))
{
	vR[i] = 1373;
}
else
{
	vR[i] = 1372;
}

}

}
else
{
	if(input[i]>(1369+offset0))
{
	if(input[i]==(1371+offset0))
{
	vR[i] = 1371;
}
else
{
	vR[i] = 1370;
}

}
else
{
	if(input[i]==(1369+offset0))
{
	vR[i] = 1369;
}
else
{
	vR[i] = 1368;
}

}

}

}
else
{
	if(input[i]>(1363+offset0))
{
	if(input[i]>(1365+offset0))
{
	if(input[i]==(1367+offset0))
{
	vR[i] = 1367;
}
else
{
	vR[i] = 1366;
}

}
else
{
	if(input[i]==(1365+offset0))
{
	vR[i] = 1365;
}
else
{
	vR[i] = 1364;
}

}

}
else
{
	if(input[i]>(1361+offset0))
{
	if(input[i]==(1363+offset0))
{
	vR[i] = 1363;
}
else
{
	vR[i] = 1362;
}

}
else
{
	if(input[i]==(1361+offset0))
{
	vR[i] = 1361;
}
else
{
	vR[i] = 1360;
}

}

}

}

}
else
{
	if(input[i]>(1351+offset0))
{
	if(input[i]>(1355+offset0))
{
	if(input[i]>(1357+offset0))
{
	if(input[i]==(1359+offset0))
{
	vR[i] = 1359;
}
else
{
	vR[i] = 1358;
}

}
else
{
	if(input[i]==(1357+offset0))
{
	vR[i] = 1357;
}
else
{
	vR[i] = 1356;
}

}

}
else
{
	if(input[i]>(1353+offset0))
{
	if(input[i]==(1355+offset0))
{
	vR[i] = 1355;
}
else
{
	vR[i] = 1354;
}

}
else
{
	if(input[i]==(1353+offset0))
{
	vR[i] = 1353;
}
else
{
	vR[i] = 1352;
}

}

}

}
else
{
	if(input[i]>(1347+offset0))
{
	if(input[i]>(1349+offset0))
{
	if(input[i]==(1351+offset0))
{
	vR[i] = 1351;
}
else
{
	vR[i] = 1350;
}

}
else
{
	if(input[i]==(1349+offset0))
{
	vR[i] = 1349;
}
else
{
	vR[i] = 1348;
}

}

}
else
{
	if(input[i]>(1345+offset0))
{
	if(input[i]==(1347+offset0))
{
	vR[i] = 1347;
}
else
{
	vR[i] = 1346;
}

}
else
{
	if(input[i]==(1345+offset0))
{
	vR[i] = 1345;
}
else
{
	vR[i] = 1344;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1311+offset0))
{
	if(input[i]>(1327+offset0))
{
	if(input[i]>(1335+offset0))
{
	if(input[i]>(1339+offset0))
{
	if(input[i]>(1341+offset0))
{
	if(input[i]==(1343+offset0))
{
	vR[i] = 1343;
}
else
{
	vR[i] = 1342;
}

}
else
{
	if(input[i]==(1341+offset0))
{
	vR[i] = 1341;
}
else
{
	vR[i] = 1340;
}

}

}
else
{
	if(input[i]>(1337+offset0))
{
	if(input[i]==(1339+offset0))
{
	vR[i] = 1339;
}
else
{
	vR[i] = 1338;
}

}
else
{
	if(input[i]==(1337+offset0))
{
	vR[i] = 1337;
}
else
{
	vR[i] = 1336;
}

}

}

}
else
{
	if(input[i]>(1331+offset0))
{
	if(input[i]>(1333+offset0))
{
	if(input[i]==(1335+offset0))
{
	vR[i] = 1335;
}
else
{
	vR[i] = 1334;
}

}
else
{
	if(input[i]==(1333+offset0))
{
	vR[i] = 1333;
}
else
{
	vR[i] = 1332;
}

}

}
else
{
	if(input[i]>(1329+offset0))
{
	if(input[i]==(1331+offset0))
{
	vR[i] = 1331;
}
else
{
	vR[i] = 1330;
}

}
else
{
	if(input[i]==(1329+offset0))
{
	vR[i] = 1329;
}
else
{
	vR[i] = 1328;
}

}

}

}

}
else
{
	if(input[i]>(1319+offset0))
{
	if(input[i]>(1323+offset0))
{
	if(input[i]>(1325+offset0))
{
	if(input[i]==(1327+offset0))
{
	vR[i] = 1327;
}
else
{
	vR[i] = 1326;
}

}
else
{
	if(input[i]==(1325+offset0))
{
	vR[i] = 1325;
}
else
{
	vR[i] = 1324;
}

}

}
else
{
	if(input[i]>(1321+offset0))
{
	if(input[i]==(1323+offset0))
{
	vR[i] = 1323;
}
else
{
	vR[i] = 1322;
}

}
else
{
	if(input[i]==(1321+offset0))
{
	vR[i] = 1321;
}
else
{
	vR[i] = 1320;
}

}

}

}
else
{
	if(input[i]>(1315+offset0))
{
	if(input[i]>(1317+offset0))
{
	if(input[i]==(1319+offset0))
{
	vR[i] = 1319;
}
else
{
	vR[i] = 1318;
}

}
else
{
	if(input[i]==(1317+offset0))
{
	vR[i] = 1317;
}
else
{
	vR[i] = 1316;
}

}

}
else
{
	if(input[i]>(1313+offset0))
{
	if(input[i]==(1315+offset0))
{
	vR[i] = 1315;
}
else
{
	vR[i] = 1314;
}

}
else
{
	if(input[i]==(1313+offset0))
{
	vR[i] = 1313;
}
else
{
	vR[i] = 1312;
}

}

}

}

}

}
else
{
	if(input[i]>(1295+offset0))
{
	if(input[i]>(1303+offset0))
{
	if(input[i]>(1307+offset0))
{
	if(input[i]>(1309+offset0))
{
	if(input[i]==(1311+offset0))
{
	vR[i] = 1311;
}
else
{
	vR[i] = 1310;
}

}
else
{
	if(input[i]==(1309+offset0))
{
	vR[i] = 1309;
}
else
{
	vR[i] = 1308;
}

}

}
else
{
	if(input[i]>(1305+offset0))
{
	if(input[i]==(1307+offset0))
{
	vR[i] = 1307;
}
else
{
	vR[i] = 1306;
}

}
else
{
	if(input[i]==(1305+offset0))
{
	vR[i] = 1305;
}
else
{
	vR[i] = 1304;
}

}

}

}
else
{
	if(input[i]>(1299+offset0))
{
	if(input[i]>(1301+offset0))
{
	if(input[i]==(1303+offset0))
{
	vR[i] = 1303;
}
else
{
	vR[i] = 1302;
}

}
else
{
	if(input[i]==(1301+offset0))
{
	vR[i] = 1301;
}
else
{
	vR[i] = 1300;
}

}

}
else
{
	if(input[i]>(1297+offset0))
{
	if(input[i]==(1299+offset0))
{
	vR[i] = 1299;
}
else
{
	vR[i] = 1298;
}

}
else
{
	if(input[i]==(1297+offset0))
{
	vR[i] = 1297;
}
else
{
	vR[i] = 1296;
}

}

}

}

}
else
{
	if(input[i]>(1287+offset0))
{
	if(input[i]>(1291+offset0))
{
	if(input[i]>(1293+offset0))
{
	if(input[i]==(1295+offset0))
{
	vR[i] = 1295;
}
else
{
	vR[i] = 1294;
}

}
else
{
	if(input[i]==(1293+offset0))
{
	vR[i] = 1293;
}
else
{
	vR[i] = 1292;
}

}

}
else
{
	if(input[i]>(1289+offset0))
{
	if(input[i]==(1291+offset0))
{
	vR[i] = 1291;
}
else
{
	vR[i] = 1290;
}

}
else
{
	if(input[i]==(1289+offset0))
{
	vR[i] = 1289;
}
else
{
	vR[i] = 1288;
}

}

}

}
else
{
	if(input[i]>(1283+offset0))
{
	if(input[i]>(1285+offset0))
{
	if(input[i]==(1287+offset0))
{
	vR[i] = 1287;
}
else
{
	vR[i] = 1286;
}

}
else
{
	if(input[i]==(1285+offset0))
{
	vR[i] = 1285;
}
else
{
	vR[i] = 1284;
}

}

}
else
{
	if(input[i]>(1281+offset0))
{
	if(input[i]==(1283+offset0))
{
	vR[i] = 1283;
}
else
{
	vR[i] = 1282;
}

}
else
{
	if(input[i]==(1281+offset0))
{
	vR[i] = 1281;
}
else
{
	vR[i] = 1280;
}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(1151+offset0))
{
	if(input[i]>(1215+offset0))
{
	if(input[i]>(1247+offset0))
{
	if(input[i]>(1263+offset0))
{
	if(input[i]>(1271+offset0))
{
	if(input[i]>(1275+offset0))
{
	if(input[i]>(1277+offset0))
{
	if(input[i]==(1279+offset0))
{
	vR[i] = 1279;
}
else
{
	vR[i] = 1278;
}

}
else
{
	if(input[i]==(1277+offset0))
{
	vR[i] = 1277;
}
else
{
	vR[i] = 1276;
}

}

}
else
{
	if(input[i]>(1273+offset0))
{
	if(input[i]==(1275+offset0))
{
	vR[i] = 1275;
}
else
{
	vR[i] = 1274;
}

}
else
{
	if(input[i]==(1273+offset0))
{
	vR[i] = 1273;
}
else
{
	vR[i] = 1272;
}

}

}

}
else
{
	if(input[i]>(1267+offset0))
{
	if(input[i]>(1269+offset0))
{
	if(input[i]==(1271+offset0))
{
	vR[i] = 1271;
}
else
{
	vR[i] = 1270;
}

}
else
{
	if(input[i]==(1269+offset0))
{
	vR[i] = 1269;
}
else
{
	vR[i] = 1268;
}

}

}
else
{
	if(input[i]>(1265+offset0))
{
	if(input[i]==(1267+offset0))
{
	vR[i] = 1267;
}
else
{
	vR[i] = 1266;
}

}
else
{
	if(input[i]==(1265+offset0))
{
	vR[i] = 1265;
}
else
{
	vR[i] = 1264;
}

}

}

}

}
else
{
	if(input[i]>(1255+offset0))
{
	if(input[i]>(1259+offset0))
{
	if(input[i]>(1261+offset0))
{
	if(input[i]==(1263+offset0))
{
	vR[i] = 1263;
}
else
{
	vR[i] = 1262;
}

}
else
{
	if(input[i]==(1261+offset0))
{
	vR[i] = 1261;
}
else
{
	vR[i] = 1260;
}

}

}
else
{
	if(input[i]>(1257+offset0))
{
	if(input[i]==(1259+offset0))
{
	vR[i] = 1259;
}
else
{
	vR[i] = 1258;
}

}
else
{
	if(input[i]==(1257+offset0))
{
	vR[i] = 1257;
}
else
{
	vR[i] = 1256;
}

}

}

}
else
{
	if(input[i]>(1251+offset0))
{
	if(input[i]>(1253+offset0))
{
	if(input[i]==(1255+offset0))
{
	vR[i] = 1255;
}
else
{
	vR[i] = 1254;
}

}
else
{
	if(input[i]==(1253+offset0))
{
	vR[i] = 1253;
}
else
{
	vR[i] = 1252;
}

}

}
else
{
	if(input[i]>(1249+offset0))
{
	if(input[i]==(1251+offset0))
{
	vR[i] = 1251;
}
else
{
	vR[i] = 1250;
}

}
else
{
	if(input[i]==(1249+offset0))
{
	vR[i] = 1249;
}
else
{
	vR[i] = 1248;
}

}

}

}

}

}
else
{
	if(input[i]>(1231+offset0))
{
	if(input[i]>(1239+offset0))
{
	if(input[i]>(1243+offset0))
{
	if(input[i]>(1245+offset0))
{
	if(input[i]==(1247+offset0))
{
	vR[i] = 1247;
}
else
{
	vR[i] = 1246;
}

}
else
{
	if(input[i]==(1245+offset0))
{
	vR[i] = 1245;
}
else
{
	vR[i] = 1244;
}

}

}
else
{
	if(input[i]>(1241+offset0))
{
	if(input[i]==(1243+offset0))
{
	vR[i] = 1243;
}
else
{
	vR[i] = 1242;
}

}
else
{
	if(input[i]==(1241+offset0))
{
	vR[i] = 1241;
}
else
{
	vR[i] = 1240;
}

}

}

}
else
{
	if(input[i]>(1235+offset0))
{
	if(input[i]>(1237+offset0))
{
	if(input[i]==(1239+offset0))
{
	vR[i] = 1239;
}
else
{
	vR[i] = 1238;
}

}
else
{
	if(input[i]==(1237+offset0))
{
	vR[i] = 1237;
}
else
{
	vR[i] = 1236;
}

}

}
else
{
	if(input[i]>(1233+offset0))
{
	if(input[i]==(1235+offset0))
{
	vR[i] = 1235;
}
else
{
	vR[i] = 1234;
}

}
else
{
	if(input[i]==(1233+offset0))
{
	vR[i] = 1233;
}
else
{
	vR[i] = 1232;
}

}

}

}

}
else
{
	if(input[i]>(1223+offset0))
{
	if(input[i]>(1227+offset0))
{
	if(input[i]>(1229+offset0))
{
	if(input[i]==(1231+offset0))
{
	vR[i] = 1231;
}
else
{
	vR[i] = 1230;
}

}
else
{
	if(input[i]==(1229+offset0))
{
	vR[i] = 1229;
}
else
{
	vR[i] = 1228;
}

}

}
else
{
	if(input[i]>(1225+offset0))
{
	if(input[i]==(1227+offset0))
{
	vR[i] = 1227;
}
else
{
	vR[i] = 1226;
}

}
else
{
	if(input[i]==(1225+offset0))
{
	vR[i] = 1225;
}
else
{
	vR[i] = 1224;
}

}

}

}
else
{
	if(input[i]>(1219+offset0))
{
	if(input[i]>(1221+offset0))
{
	if(input[i]==(1223+offset0))
{
	vR[i] = 1223;
}
else
{
	vR[i] = 1222;
}

}
else
{
	if(input[i]==(1221+offset0))
{
	vR[i] = 1221;
}
else
{
	vR[i] = 1220;
}

}

}
else
{
	if(input[i]>(1217+offset0))
{
	if(input[i]==(1219+offset0))
{
	vR[i] = 1219;
}
else
{
	vR[i] = 1218;
}

}
else
{
	if(input[i]==(1217+offset0))
{
	vR[i] = 1217;
}
else
{
	vR[i] = 1216;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1183+offset0))
{
	if(input[i]>(1199+offset0))
{
	if(input[i]>(1207+offset0))
{
	if(input[i]>(1211+offset0))
{
	if(input[i]>(1213+offset0))
{
	if(input[i]==(1215+offset0))
{
	vR[i] = 1215;
}
else
{
	vR[i] = 1214;
}

}
else
{
	if(input[i]==(1213+offset0))
{
	vR[i] = 1213;
}
else
{
	vR[i] = 1212;
}

}

}
else
{
	if(input[i]>(1209+offset0))
{
	if(input[i]==(1211+offset0))
{
	vR[i] = 1211;
}
else
{
	vR[i] = 1210;
}

}
else
{
	if(input[i]==(1209+offset0))
{
	vR[i] = 1209;
}
else
{
	vR[i] = 1208;
}

}

}

}
else
{
	if(input[i]>(1203+offset0))
{
	if(input[i]>(1205+offset0))
{
	if(input[i]==(1207+offset0))
{
	vR[i] = 1207;
}
else
{
	vR[i] = 1206;
}

}
else
{
	if(input[i]==(1205+offset0))
{
	vR[i] = 1205;
}
else
{
	vR[i] = 1204;
}

}

}
else
{
	if(input[i]>(1201+offset0))
{
	if(input[i]==(1203+offset0))
{
	vR[i] = 1203;
}
else
{
	vR[i] = 1202;
}

}
else
{
	if(input[i]==(1201+offset0))
{
	vR[i] = 1201;
}
else
{
	vR[i] = 1200;
}

}

}

}

}
else
{
	if(input[i]>(1191+offset0))
{
	if(input[i]>(1195+offset0))
{
	if(input[i]>(1197+offset0))
{
	if(input[i]==(1199+offset0))
{
	vR[i] = 1199;
}
else
{
	vR[i] = 1198;
}

}
else
{
	if(input[i]==(1197+offset0))
{
	vR[i] = 1197;
}
else
{
	vR[i] = 1196;
}

}

}
else
{
	if(input[i]>(1193+offset0))
{
	if(input[i]==(1195+offset0))
{
	vR[i] = 1195;
}
else
{
	vR[i] = 1194;
}

}
else
{
	if(input[i]==(1193+offset0))
{
	vR[i] = 1193;
}
else
{
	vR[i] = 1192;
}

}

}

}
else
{
	if(input[i]>(1187+offset0))
{
	if(input[i]>(1189+offset0))
{
	if(input[i]==(1191+offset0))
{
	vR[i] = 1191;
}
else
{
	vR[i] = 1190;
}

}
else
{
	if(input[i]==(1189+offset0))
{
	vR[i] = 1189;
}
else
{
	vR[i] = 1188;
}

}

}
else
{
	if(input[i]>(1185+offset0))
{
	if(input[i]==(1187+offset0))
{
	vR[i] = 1187;
}
else
{
	vR[i] = 1186;
}

}
else
{
	if(input[i]==(1185+offset0))
{
	vR[i] = 1185;
}
else
{
	vR[i] = 1184;
}

}

}

}

}

}
else
{
	if(input[i]>(1167+offset0))
{
	if(input[i]>(1175+offset0))
{
	if(input[i]>(1179+offset0))
{
	if(input[i]>(1181+offset0))
{
	if(input[i]==(1183+offset0))
{
	vR[i] = 1183;
}
else
{
	vR[i] = 1182;
}

}
else
{
	if(input[i]==(1181+offset0))
{
	vR[i] = 1181;
}
else
{
	vR[i] = 1180;
}

}

}
else
{
	if(input[i]>(1177+offset0))
{
	if(input[i]==(1179+offset0))
{
	vR[i] = 1179;
}
else
{
	vR[i] = 1178;
}

}
else
{
	if(input[i]==(1177+offset0))
{
	vR[i] = 1177;
}
else
{
	vR[i] = 1176;
}

}

}

}
else
{
	if(input[i]>(1171+offset0))
{
	if(input[i]>(1173+offset0))
{
	if(input[i]==(1175+offset0))
{
	vR[i] = 1175;
}
else
{
	vR[i] = 1174;
}

}
else
{
	if(input[i]==(1173+offset0))
{
	vR[i] = 1173;
}
else
{
	vR[i] = 1172;
}

}

}
else
{
	if(input[i]>(1169+offset0))
{
	if(input[i]==(1171+offset0))
{
	vR[i] = 1171;
}
else
{
	vR[i] = 1170;
}

}
else
{
	if(input[i]==(1169+offset0))
{
	vR[i] = 1169;
}
else
{
	vR[i] = 1168;
}

}

}

}

}
else
{
	if(input[i]>(1159+offset0))
{
	if(input[i]>(1163+offset0))
{
	if(input[i]>(1165+offset0))
{
	if(input[i]==(1167+offset0))
{
	vR[i] = 1167;
}
else
{
	vR[i] = 1166;
}

}
else
{
	if(input[i]==(1165+offset0))
{
	vR[i] = 1165;
}
else
{
	vR[i] = 1164;
}

}

}
else
{
	if(input[i]>(1161+offset0))
{
	if(input[i]==(1163+offset0))
{
	vR[i] = 1163;
}
else
{
	vR[i] = 1162;
}

}
else
{
	if(input[i]==(1161+offset0))
{
	vR[i] = 1161;
}
else
{
	vR[i] = 1160;
}

}

}

}
else
{
	if(input[i]>(1155+offset0))
{
	if(input[i]>(1157+offset0))
{
	if(input[i]==(1159+offset0))
{
	vR[i] = 1159;
}
else
{
	vR[i] = 1158;
}

}
else
{
	if(input[i]==(1157+offset0))
{
	vR[i] = 1157;
}
else
{
	vR[i] = 1156;
}

}

}
else
{
	if(input[i]>(1153+offset0))
{
	if(input[i]==(1155+offset0))
{
	vR[i] = 1155;
}
else
{
	vR[i] = 1154;
}

}
else
{
	if(input[i]==(1153+offset0))
{
	vR[i] = 1153;
}
else
{
	vR[i] = 1152;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(1087+offset0))
{
	if(input[i]>(1119+offset0))
{
	if(input[i]>(1135+offset0))
{
	if(input[i]>(1143+offset0))
{
	if(input[i]>(1147+offset0))
{
	if(input[i]>(1149+offset0))
{
	if(input[i]==(1151+offset0))
{
	vR[i] = 1151;
}
else
{
	vR[i] = 1150;
}

}
else
{
	if(input[i]==(1149+offset0))
{
	vR[i] = 1149;
}
else
{
	vR[i] = 1148;
}

}

}
else
{
	if(input[i]>(1145+offset0))
{
	if(input[i]==(1147+offset0))
{
	vR[i] = 1147;
}
else
{
	vR[i] = 1146;
}

}
else
{
	if(input[i]==(1145+offset0))
{
	vR[i] = 1145;
}
else
{
	vR[i] = 1144;
}

}

}

}
else
{
	if(input[i]>(1139+offset0))
{
	if(input[i]>(1141+offset0))
{
	if(input[i]==(1143+offset0))
{
	vR[i] = 1143;
}
else
{
	vR[i] = 1142;
}

}
else
{
	if(input[i]==(1141+offset0))
{
	vR[i] = 1141;
}
else
{
	vR[i] = 1140;
}

}

}
else
{
	if(input[i]>(1137+offset0))
{
	if(input[i]==(1139+offset0))
{
	vR[i] = 1139;
}
else
{
	vR[i] = 1138;
}

}
else
{
	if(input[i]==(1137+offset0))
{
	vR[i] = 1137;
}
else
{
	vR[i] = 1136;
}

}

}

}

}
else
{
	if(input[i]>(1127+offset0))
{
	if(input[i]>(1131+offset0))
{
	if(input[i]>(1133+offset0))
{
	if(input[i]==(1135+offset0))
{
	vR[i] = 1135;
}
else
{
	vR[i] = 1134;
}

}
else
{
	if(input[i]==(1133+offset0))
{
	vR[i] = 1133;
}
else
{
	vR[i] = 1132;
}

}

}
else
{
	if(input[i]>(1129+offset0))
{
	if(input[i]==(1131+offset0))
{
	vR[i] = 1131;
}
else
{
	vR[i] = 1130;
}

}
else
{
	if(input[i]==(1129+offset0))
{
	vR[i] = 1129;
}
else
{
	vR[i] = 1128;
}

}

}

}
else
{
	if(input[i]>(1123+offset0))
{
	if(input[i]>(1125+offset0))
{
	if(input[i]==(1127+offset0))
{
	vR[i] = 1127;
}
else
{
	vR[i] = 1126;
}

}
else
{
	if(input[i]==(1125+offset0))
{
	vR[i] = 1125;
}
else
{
	vR[i] = 1124;
}

}

}
else
{
	if(input[i]>(1121+offset0))
{
	if(input[i]==(1123+offset0))
{
	vR[i] = 1123;
}
else
{
	vR[i] = 1122;
}

}
else
{
	if(input[i]==(1121+offset0))
{
	vR[i] = 1121;
}
else
{
	vR[i] = 1120;
}

}

}

}

}

}
else
{
	if(input[i]>(1103+offset0))
{
	if(input[i]>(1111+offset0))
{
	if(input[i]>(1115+offset0))
{
	if(input[i]>(1117+offset0))
{
	if(input[i]==(1119+offset0))
{
	vR[i] = 1119;
}
else
{
	vR[i] = 1118;
}

}
else
{
	if(input[i]==(1117+offset0))
{
	vR[i] = 1117;
}
else
{
	vR[i] = 1116;
}

}

}
else
{
	if(input[i]>(1113+offset0))
{
	if(input[i]==(1115+offset0))
{
	vR[i] = 1115;
}
else
{
	vR[i] = 1114;
}

}
else
{
	if(input[i]==(1113+offset0))
{
	vR[i] = 1113;
}
else
{
	vR[i] = 1112;
}

}

}

}
else
{
	if(input[i]>(1107+offset0))
{
	if(input[i]>(1109+offset0))
{
	if(input[i]==(1111+offset0))
{
	vR[i] = 1111;
}
else
{
	vR[i] = 1110;
}

}
else
{
	if(input[i]==(1109+offset0))
{
	vR[i] = 1109;
}
else
{
	vR[i] = 1108;
}

}

}
else
{
	if(input[i]>(1105+offset0))
{
	if(input[i]==(1107+offset0))
{
	vR[i] = 1107;
}
else
{
	vR[i] = 1106;
}

}
else
{
	if(input[i]==(1105+offset0))
{
	vR[i] = 1105;
}
else
{
	vR[i] = 1104;
}

}

}

}

}
else
{
	if(input[i]>(1095+offset0))
{
	if(input[i]>(1099+offset0))
{
	if(input[i]>(1101+offset0))
{
	if(input[i]==(1103+offset0))
{
	vR[i] = 1103;
}
else
{
	vR[i] = 1102;
}

}
else
{
	if(input[i]==(1101+offset0))
{
	vR[i] = 1101;
}
else
{
	vR[i] = 1100;
}

}

}
else
{
	if(input[i]>(1097+offset0))
{
	if(input[i]==(1099+offset0))
{
	vR[i] = 1099;
}
else
{
	vR[i] = 1098;
}

}
else
{
	if(input[i]==(1097+offset0))
{
	vR[i] = 1097;
}
else
{
	vR[i] = 1096;
}

}

}

}
else
{
	if(input[i]>(1091+offset0))
{
	if(input[i]>(1093+offset0))
{
	if(input[i]==(1095+offset0))
{
	vR[i] = 1095;
}
else
{
	vR[i] = 1094;
}

}
else
{
	if(input[i]==(1093+offset0))
{
	vR[i] = 1093;
}
else
{
	vR[i] = 1092;
}

}

}
else
{
	if(input[i]>(1089+offset0))
{
	if(input[i]==(1091+offset0))
{
	vR[i] = 1091;
}
else
{
	vR[i] = 1090;
}

}
else
{
	if(input[i]==(1089+offset0))
{
	vR[i] = 1089;
}
else
{
	vR[i] = 1088;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1055+offset0))
{
	if(input[i]>(1071+offset0))
{
	if(input[i]>(1079+offset0))
{
	if(input[i]>(1083+offset0))
{
	if(input[i]>(1085+offset0))
{
	if(input[i]==(1087+offset0))
{
	vR[i] = 1087;
}
else
{
	vR[i] = 1086;
}

}
else
{
	if(input[i]==(1085+offset0))
{
	vR[i] = 1085;
}
else
{
	vR[i] = 1084;
}

}

}
else
{
	if(input[i]>(1081+offset0))
{
	if(input[i]==(1083+offset0))
{
	vR[i] = 1083;
}
else
{
	vR[i] = 1082;
}

}
else
{
	if(input[i]==(1081+offset0))
{
	vR[i] = 1081;
}
else
{
	vR[i] = 1080;
}

}

}

}
else
{
	if(input[i]>(1075+offset0))
{
	if(input[i]>(1077+offset0))
{
	if(input[i]==(1079+offset0))
{
	vR[i] = 1079;
}
else
{
	vR[i] = 1078;
}

}
else
{
	if(input[i]==(1077+offset0))
{
	vR[i] = 1077;
}
else
{
	vR[i] = 1076;
}

}

}
else
{
	if(input[i]>(1073+offset0))
{
	if(input[i]==(1075+offset0))
{
	vR[i] = 1075;
}
else
{
	vR[i] = 1074;
}

}
else
{
	if(input[i]==(1073+offset0))
{
	vR[i] = 1073;
}
else
{
	vR[i] = 1072;
}

}

}

}

}
else
{
	if(input[i]>(1063+offset0))
{
	if(input[i]>(1067+offset0))
{
	if(input[i]>(1069+offset0))
{
	if(input[i]==(1071+offset0))
{
	vR[i] = 1071;
}
else
{
	vR[i] = 1070;
}

}
else
{
	if(input[i]==(1069+offset0))
{
	vR[i] = 1069;
}
else
{
	vR[i] = 1068;
}

}

}
else
{
	if(input[i]>(1065+offset0))
{
	if(input[i]==(1067+offset0))
{
	vR[i] = 1067;
}
else
{
	vR[i] = 1066;
}

}
else
{
	if(input[i]==(1065+offset0))
{
	vR[i] = 1065;
}
else
{
	vR[i] = 1064;
}

}

}

}
else
{
	if(input[i]>(1059+offset0))
{
	if(input[i]>(1061+offset0))
{
	if(input[i]==(1063+offset0))
{
	vR[i] = 1063;
}
else
{
	vR[i] = 1062;
}

}
else
{
	if(input[i]==(1061+offset0))
{
	vR[i] = 1061;
}
else
{
	vR[i] = 1060;
}

}

}
else
{
	if(input[i]>(1057+offset0))
{
	if(input[i]==(1059+offset0))
{
	vR[i] = 1059;
}
else
{
	vR[i] = 1058;
}

}
else
{
	if(input[i]==(1057+offset0))
{
	vR[i] = 1057;
}
else
{
	vR[i] = 1056;
}

}

}

}

}

}
else
{
	if(input[i]>(1039+offset0))
{
	if(input[i]>(1047+offset0))
{
	if(input[i]>(1051+offset0))
{
	if(input[i]>(1053+offset0))
{
	if(input[i]==(1055+offset0))
{
	vR[i] = 1055;
}
else
{
	vR[i] = 1054;
}

}
else
{
	if(input[i]==(1053+offset0))
{
	vR[i] = 1053;
}
else
{
	vR[i] = 1052;
}

}

}
else
{
	if(input[i]>(1049+offset0))
{
	if(input[i]==(1051+offset0))
{
	vR[i] = 1051;
}
else
{
	vR[i] = 1050;
}

}
else
{
	if(input[i]==(1049+offset0))
{
	vR[i] = 1049;
}
else
{
	vR[i] = 1048;
}

}

}

}
else
{
	if(input[i]>(1043+offset0))
{
	if(input[i]>(1045+offset0))
{
	if(input[i]==(1047+offset0))
{
	vR[i] = 1047;
}
else
{
	vR[i] = 1046;
}

}
else
{
	if(input[i]==(1045+offset0))
{
	vR[i] = 1045;
}
else
{
	vR[i] = 1044;
}

}

}
else
{
	if(input[i]>(1041+offset0))
{
	if(input[i]==(1043+offset0))
{
	vR[i] = 1043;
}
else
{
	vR[i] = 1042;
}

}
else
{
	if(input[i]==(1041+offset0))
{
	vR[i] = 1041;
}
else
{
	vR[i] = 1040;
}

}

}

}

}
else
{
	if(input[i]>(1031+offset0))
{
	if(input[i]>(1035+offset0))
{
	if(input[i]>(1037+offset0))
{
	if(input[i]==(1039+offset0))
{
	vR[i] = 1039;
}
else
{
	vR[i] = 1038;
}

}
else
{
	if(input[i]==(1037+offset0))
{
	vR[i] = 1037;
}
else
{
	vR[i] = 1036;
}

}

}
else
{
	if(input[i]>(1033+offset0))
{
	if(input[i]==(1035+offset0))
{
	vR[i] = 1035;
}
else
{
	vR[i] = 1034;
}

}
else
{
	if(input[i]==(1033+offset0))
{
	vR[i] = 1033;
}
else
{
	vR[i] = 1032;
}

}

}

}
else
{
	if(input[i]>(1027+offset0))
{
	if(input[i]>(1029+offset0))
{
	if(input[i]==(1031+offset0))
{
	vR[i] = 1031;
}
else
{
	vR[i] = 1030;
}

}
else
{
	if(input[i]==(1029+offset0))
{
	vR[i] = 1029;
}
else
{
	vR[i] = 1028;
}

}

}
else
{
	if(input[i]>(1025+offset0))
{
	if(input[i]==(1027+offset0))
{
	vR[i] = 1027;
}
else
{
	vR[i] = 1026;
}

}
else
{
	if(input[i]==(1025+offset0))
{
	vR[i] = 1025;
}
else
{
	vR[i] = 1024;
}

}

}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(511+offset0))
{
	if(input[i]>(767+offset0))
{
	if(input[i]>(895+offset0))
{
	if(input[i]>(959+offset0))
{
	if(input[i]>(991+offset0))
{
	if(input[i]>(1007+offset0))
{
	if(input[i]>(1015+offset0))
{
	if(input[i]>(1019+offset0))
{
	if(input[i]>(1021+offset0))
{
	if(input[i]==(1023+offset0))
{
	vR[i] = 1023;
}
else
{
	vR[i] = 1022;
}

}
else
{
	if(input[i]==(1021+offset0))
{
	vR[i] = 1021;
}
else
{
	vR[i] = 1020;
}

}

}
else
{
	if(input[i]>(1017+offset0))
{
	if(input[i]==(1019+offset0))
{
	vR[i] = 1019;
}
else
{
	vR[i] = 1018;
}

}
else
{
	if(input[i]==(1017+offset0))
{
	vR[i] = 1017;
}
else
{
	vR[i] = 1016;
}

}

}

}
else
{
	if(input[i]>(1011+offset0))
{
	if(input[i]>(1013+offset0))
{
	if(input[i]==(1015+offset0))
{
	vR[i] = 1015;
}
else
{
	vR[i] = 1014;
}

}
else
{
	if(input[i]==(1013+offset0))
{
	vR[i] = 1013;
}
else
{
	vR[i] = 1012;
}

}

}
else
{
	if(input[i]>(1009+offset0))
{
	if(input[i]==(1011+offset0))
{
	vR[i] = 1011;
}
else
{
	vR[i] = 1010;
}

}
else
{
	if(input[i]==(1009+offset0))
{
	vR[i] = 1009;
}
else
{
	vR[i] = 1008;
}

}

}

}

}
else
{
	if(input[i]>(999+offset0))
{
	if(input[i]>(1003+offset0))
{
	if(input[i]>(1005+offset0))
{
	if(input[i]==(1007+offset0))
{
	vR[i] = 1007;
}
else
{
	vR[i] = 1006;
}

}
else
{
	if(input[i]==(1005+offset0))
{
	vR[i] = 1005;
}
else
{
	vR[i] = 1004;
}

}

}
else
{
	if(input[i]>(1001+offset0))
{
	if(input[i]==(1003+offset0))
{
	vR[i] = 1003;
}
else
{
	vR[i] = 1002;
}

}
else
{
	if(input[i]==(1001+offset0))
{
	vR[i] = 1001;
}
else
{
	vR[i] = 1000;
}

}

}

}
else
{
	if(input[i]>(995+offset0))
{
	if(input[i]>(997+offset0))
{
	if(input[i]==(999+offset0))
{
	vR[i] = 999;
}
else
{
	vR[i] = 998;
}

}
else
{
	if(input[i]==(997+offset0))
{
	vR[i] = 997;
}
else
{
	vR[i] = 996;
}

}

}
else
{
	if(input[i]>(993+offset0))
{
	if(input[i]==(995+offset0))
{
	vR[i] = 995;
}
else
{
	vR[i] = 994;
}

}
else
{
	if(input[i]==(993+offset0))
{
	vR[i] = 993;
}
else
{
	vR[i] = 992;
}

}

}

}

}

}
else
{
	if(input[i]>(975+offset0))
{
	if(input[i]>(983+offset0))
{
	if(input[i]>(987+offset0))
{
	if(input[i]>(989+offset0))
{
	if(input[i]==(991+offset0))
{
	vR[i] = 991;
}
else
{
	vR[i] = 990;
}

}
else
{
	if(input[i]==(989+offset0))
{
	vR[i] = 989;
}
else
{
	vR[i] = 988;
}

}

}
else
{
	if(input[i]>(985+offset0))
{
	if(input[i]==(987+offset0))
{
	vR[i] = 987;
}
else
{
	vR[i] = 986;
}

}
else
{
	if(input[i]==(985+offset0))
{
	vR[i] = 985;
}
else
{
	vR[i] = 984;
}

}

}

}
else
{
	if(input[i]>(979+offset0))
{
	if(input[i]>(981+offset0))
{
	if(input[i]==(983+offset0))
{
	vR[i] = 983;
}
else
{
	vR[i] = 982;
}

}
else
{
	if(input[i]==(981+offset0))
{
	vR[i] = 981;
}
else
{
	vR[i] = 980;
}

}

}
else
{
	if(input[i]>(977+offset0))
{
	if(input[i]==(979+offset0))
{
	vR[i] = 979;
}
else
{
	vR[i] = 978;
}

}
else
{
	if(input[i]==(977+offset0))
{
	vR[i] = 977;
}
else
{
	vR[i] = 976;
}

}

}

}

}
else
{
	if(input[i]>(967+offset0))
{
	if(input[i]>(971+offset0))
{
	if(input[i]>(973+offset0))
{
	if(input[i]==(975+offset0))
{
	vR[i] = 975;
}
else
{
	vR[i] = 974;
}

}
else
{
	if(input[i]==(973+offset0))
{
	vR[i] = 973;
}
else
{
	vR[i] = 972;
}

}

}
else
{
	if(input[i]>(969+offset0))
{
	if(input[i]==(971+offset0))
{
	vR[i] = 971;
}
else
{
	vR[i] = 970;
}

}
else
{
	if(input[i]==(969+offset0))
{
	vR[i] = 969;
}
else
{
	vR[i] = 968;
}

}

}

}
else
{
	if(input[i]>(963+offset0))
{
	if(input[i]>(965+offset0))
{
	if(input[i]==(967+offset0))
{
	vR[i] = 967;
}
else
{
	vR[i] = 966;
}

}
else
{
	if(input[i]==(965+offset0))
{
	vR[i] = 965;
}
else
{
	vR[i] = 964;
}

}

}
else
{
	if(input[i]>(961+offset0))
{
	if(input[i]==(963+offset0))
{
	vR[i] = 963;
}
else
{
	vR[i] = 962;
}

}
else
{
	if(input[i]==(961+offset0))
{
	vR[i] = 961;
}
else
{
	vR[i] = 960;
}

}

}

}

}

}

}
else
{
	if(input[i]>(927+offset0))
{
	if(input[i]>(943+offset0))
{
	if(input[i]>(951+offset0))
{
	if(input[i]>(955+offset0))
{
	if(input[i]>(957+offset0))
{
	if(input[i]==(959+offset0))
{
	vR[i] = 959;
}
else
{
	vR[i] = 958;
}

}
else
{
	if(input[i]==(957+offset0))
{
	vR[i] = 957;
}
else
{
	vR[i] = 956;
}

}

}
else
{
	if(input[i]>(953+offset0))
{
	if(input[i]==(955+offset0))
{
	vR[i] = 955;
}
else
{
	vR[i] = 954;
}

}
else
{
	if(input[i]==(953+offset0))
{
	vR[i] = 953;
}
else
{
	vR[i] = 952;
}

}

}

}
else
{
	if(input[i]>(947+offset0))
{
	if(input[i]>(949+offset0))
{
	if(input[i]==(951+offset0))
{
	vR[i] = 951;
}
else
{
	vR[i] = 950;
}

}
else
{
	if(input[i]==(949+offset0))
{
	vR[i] = 949;
}
else
{
	vR[i] = 948;
}

}

}
else
{
	if(input[i]>(945+offset0))
{
	if(input[i]==(947+offset0))
{
	vR[i] = 947;
}
else
{
	vR[i] = 946;
}

}
else
{
	if(input[i]==(945+offset0))
{
	vR[i] = 945;
}
else
{
	vR[i] = 944;
}

}

}

}

}
else
{
	if(input[i]>(935+offset0))
{
	if(input[i]>(939+offset0))
{
	if(input[i]>(941+offset0))
{
	if(input[i]==(943+offset0))
{
	vR[i] = 943;
}
else
{
	vR[i] = 942;
}

}
else
{
	if(input[i]==(941+offset0))
{
	vR[i] = 941;
}
else
{
	vR[i] = 940;
}

}

}
else
{
	if(input[i]>(937+offset0))
{
	if(input[i]==(939+offset0))
{
	vR[i] = 939;
}
else
{
	vR[i] = 938;
}

}
else
{
	if(input[i]==(937+offset0))
{
	vR[i] = 937;
}
else
{
	vR[i] = 936;
}

}

}

}
else
{
	if(input[i]>(931+offset0))
{
	if(input[i]>(933+offset0))
{
	if(input[i]==(935+offset0))
{
	vR[i] = 935;
}
else
{
	vR[i] = 934;
}

}
else
{
	if(input[i]==(933+offset0))
{
	vR[i] = 933;
}
else
{
	vR[i] = 932;
}

}

}
else
{
	if(input[i]>(929+offset0))
{
	if(input[i]==(931+offset0))
{
	vR[i] = 931;
}
else
{
	vR[i] = 930;
}

}
else
{
	if(input[i]==(929+offset0))
{
	vR[i] = 929;
}
else
{
	vR[i] = 928;
}

}

}

}

}

}
else
{
	if(input[i]>(911+offset0))
{
	if(input[i]>(919+offset0))
{
	if(input[i]>(923+offset0))
{
	if(input[i]>(925+offset0))
{
	if(input[i]==(927+offset0))
{
	vR[i] = 927;
}
else
{
	vR[i] = 926;
}

}
else
{
	if(input[i]==(925+offset0))
{
	vR[i] = 925;
}
else
{
	vR[i] = 924;
}

}

}
else
{
	if(input[i]>(921+offset0))
{
	if(input[i]==(923+offset0))
{
	vR[i] = 923;
}
else
{
	vR[i] = 922;
}

}
else
{
	if(input[i]==(921+offset0))
{
	vR[i] = 921;
}
else
{
	vR[i] = 920;
}

}

}

}
else
{
	if(input[i]>(915+offset0))
{
	if(input[i]>(917+offset0))
{
	if(input[i]==(919+offset0))
{
	vR[i] = 919;
}
else
{
	vR[i] = 918;
}

}
else
{
	if(input[i]==(917+offset0))
{
	vR[i] = 917;
}
else
{
	vR[i] = 916;
}

}

}
else
{
	if(input[i]>(913+offset0))
{
	if(input[i]==(915+offset0))
{
	vR[i] = 915;
}
else
{
	vR[i] = 914;
}

}
else
{
	if(input[i]==(913+offset0))
{
	vR[i] = 913;
}
else
{
	vR[i] = 912;
}

}

}

}

}
else
{
	if(input[i]>(903+offset0))
{
	if(input[i]>(907+offset0))
{
	if(input[i]>(909+offset0))
{
	if(input[i]==(911+offset0))
{
	vR[i] = 911;
}
else
{
	vR[i] = 910;
}

}
else
{
	if(input[i]==(909+offset0))
{
	vR[i] = 909;
}
else
{
	vR[i] = 908;
}

}

}
else
{
	if(input[i]>(905+offset0))
{
	if(input[i]==(907+offset0))
{
	vR[i] = 907;
}
else
{
	vR[i] = 906;
}

}
else
{
	if(input[i]==(905+offset0))
{
	vR[i] = 905;
}
else
{
	vR[i] = 904;
}

}

}

}
else
{
	if(input[i]>(899+offset0))
{
	if(input[i]>(901+offset0))
{
	if(input[i]==(903+offset0))
{
	vR[i] = 903;
}
else
{
	vR[i] = 902;
}

}
else
{
	if(input[i]==(901+offset0))
{
	vR[i] = 901;
}
else
{
	vR[i] = 900;
}

}

}
else
{
	if(input[i]>(897+offset0))
{
	if(input[i]==(899+offset0))
{
	vR[i] = 899;
}
else
{
	vR[i] = 898;
}

}
else
{
	if(input[i]==(897+offset0))
{
	vR[i] = 897;
}
else
{
	vR[i] = 896;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(831+offset0))
{
	if(input[i]>(863+offset0))
{
	if(input[i]>(879+offset0))
{
	if(input[i]>(887+offset0))
{
	if(input[i]>(891+offset0))
{
	if(input[i]>(893+offset0))
{
	if(input[i]==(895+offset0))
{
	vR[i] = 895;
}
else
{
	vR[i] = 894;
}

}
else
{
	if(input[i]==(893+offset0))
{
	vR[i] = 893;
}
else
{
	vR[i] = 892;
}

}

}
else
{
	if(input[i]>(889+offset0))
{
	if(input[i]==(891+offset0))
{
	vR[i] = 891;
}
else
{
	vR[i] = 890;
}

}
else
{
	if(input[i]==(889+offset0))
{
	vR[i] = 889;
}
else
{
	vR[i] = 888;
}

}

}

}
else
{
	if(input[i]>(883+offset0))
{
	if(input[i]>(885+offset0))
{
	if(input[i]==(887+offset0))
{
	vR[i] = 887;
}
else
{
	vR[i] = 886;
}

}
else
{
	if(input[i]==(885+offset0))
{
	vR[i] = 885;
}
else
{
	vR[i] = 884;
}

}

}
else
{
	if(input[i]>(881+offset0))
{
	if(input[i]==(883+offset0))
{
	vR[i] = 883;
}
else
{
	vR[i] = 882;
}

}
else
{
	if(input[i]==(881+offset0))
{
	vR[i] = 881;
}
else
{
	vR[i] = 880;
}

}

}

}

}
else
{
	if(input[i]>(871+offset0))
{
	if(input[i]>(875+offset0))
{
	if(input[i]>(877+offset0))
{
	if(input[i]==(879+offset0))
{
	vR[i] = 879;
}
else
{
	vR[i] = 878;
}

}
else
{
	if(input[i]==(877+offset0))
{
	vR[i] = 877;
}
else
{
	vR[i] = 876;
}

}

}
else
{
	if(input[i]>(873+offset0))
{
	if(input[i]==(875+offset0))
{
	vR[i] = 875;
}
else
{
	vR[i] = 874;
}

}
else
{
	if(input[i]==(873+offset0))
{
	vR[i] = 873;
}
else
{
	vR[i] = 872;
}

}

}

}
else
{
	if(input[i]>(867+offset0))
{
	if(input[i]>(869+offset0))
{
	if(input[i]==(871+offset0))
{
	vR[i] = 871;
}
else
{
	vR[i] = 870;
}

}
else
{
	if(input[i]==(869+offset0))
{
	vR[i] = 869;
}
else
{
	vR[i] = 868;
}

}

}
else
{
	if(input[i]>(865+offset0))
{
	if(input[i]==(867+offset0))
{
	vR[i] = 867;
}
else
{
	vR[i] = 866;
}

}
else
{
	if(input[i]==(865+offset0))
{
	vR[i] = 865;
}
else
{
	vR[i] = 864;
}

}

}

}

}

}
else
{
	if(input[i]>(847+offset0))
{
	if(input[i]>(855+offset0))
{
	if(input[i]>(859+offset0))
{
	if(input[i]>(861+offset0))
{
	if(input[i]==(863+offset0))
{
	vR[i] = 863;
}
else
{
	vR[i] = 862;
}

}
else
{
	if(input[i]==(861+offset0))
{
	vR[i] = 861;
}
else
{
	vR[i] = 860;
}

}

}
else
{
	if(input[i]>(857+offset0))
{
	if(input[i]==(859+offset0))
{
	vR[i] = 859;
}
else
{
	vR[i] = 858;
}

}
else
{
	if(input[i]==(857+offset0))
{
	vR[i] = 857;
}
else
{
	vR[i] = 856;
}

}

}

}
else
{
	if(input[i]>(851+offset0))
{
	if(input[i]>(853+offset0))
{
	if(input[i]==(855+offset0))
{
	vR[i] = 855;
}
else
{
	vR[i] = 854;
}

}
else
{
	if(input[i]==(853+offset0))
{
	vR[i] = 853;
}
else
{
	vR[i] = 852;
}

}

}
else
{
	if(input[i]>(849+offset0))
{
	if(input[i]==(851+offset0))
{
	vR[i] = 851;
}
else
{
	vR[i] = 850;
}

}
else
{
	if(input[i]==(849+offset0))
{
	vR[i] = 849;
}
else
{
	vR[i] = 848;
}

}

}

}

}
else
{
	if(input[i]>(839+offset0))
{
	if(input[i]>(843+offset0))
{
	if(input[i]>(845+offset0))
{
	if(input[i]==(847+offset0))
{
	vR[i] = 847;
}
else
{
	vR[i] = 846;
}

}
else
{
	if(input[i]==(845+offset0))
{
	vR[i] = 845;
}
else
{
	vR[i] = 844;
}

}

}
else
{
	if(input[i]>(841+offset0))
{
	if(input[i]==(843+offset0))
{
	vR[i] = 843;
}
else
{
	vR[i] = 842;
}

}
else
{
	if(input[i]==(841+offset0))
{
	vR[i] = 841;
}
else
{
	vR[i] = 840;
}

}

}

}
else
{
	if(input[i]>(835+offset0))
{
	if(input[i]>(837+offset0))
{
	if(input[i]==(839+offset0))
{
	vR[i] = 839;
}
else
{
	vR[i] = 838;
}

}
else
{
	if(input[i]==(837+offset0))
{
	vR[i] = 837;
}
else
{
	vR[i] = 836;
}

}

}
else
{
	if(input[i]>(833+offset0))
{
	if(input[i]==(835+offset0))
{
	vR[i] = 835;
}
else
{
	vR[i] = 834;
}

}
else
{
	if(input[i]==(833+offset0))
{
	vR[i] = 833;
}
else
{
	vR[i] = 832;
}

}

}

}

}

}

}
else
{
	if(input[i]>(799+offset0))
{
	if(input[i]>(815+offset0))
{
	if(input[i]>(823+offset0))
{
	if(input[i]>(827+offset0))
{
	if(input[i]>(829+offset0))
{
	if(input[i]==(831+offset0))
{
	vR[i] = 831;
}
else
{
	vR[i] = 830;
}

}
else
{
	if(input[i]==(829+offset0))
{
	vR[i] = 829;
}
else
{
	vR[i] = 828;
}

}

}
else
{
	if(input[i]>(825+offset0))
{
	if(input[i]==(827+offset0))
{
	vR[i] = 827;
}
else
{
	vR[i] = 826;
}

}
else
{
	if(input[i]==(825+offset0))
{
	vR[i] = 825;
}
else
{
	vR[i] = 824;
}

}

}

}
else
{
	if(input[i]>(819+offset0))
{
	if(input[i]>(821+offset0))
{
	if(input[i]==(823+offset0))
{
	vR[i] = 823;
}
else
{
	vR[i] = 822;
}

}
else
{
	if(input[i]==(821+offset0))
{
	vR[i] = 821;
}
else
{
	vR[i] = 820;
}

}

}
else
{
	if(input[i]>(817+offset0))
{
	if(input[i]==(819+offset0))
{
	vR[i] = 819;
}
else
{
	vR[i] = 818;
}

}
else
{
	if(input[i]==(817+offset0))
{
	vR[i] = 817;
}
else
{
	vR[i] = 816;
}

}

}

}

}
else
{
	if(input[i]>(807+offset0))
{
	if(input[i]>(811+offset0))
{
	if(input[i]>(813+offset0))
{
	if(input[i]==(815+offset0))
{
	vR[i] = 815;
}
else
{
	vR[i] = 814;
}

}
else
{
	if(input[i]==(813+offset0))
{
	vR[i] = 813;
}
else
{
	vR[i] = 812;
}

}

}
else
{
	if(input[i]>(809+offset0))
{
	if(input[i]==(811+offset0))
{
	vR[i] = 811;
}
else
{
	vR[i] = 810;
}

}
else
{
	if(input[i]==(809+offset0))
{
	vR[i] = 809;
}
else
{
	vR[i] = 808;
}

}

}

}
else
{
	if(input[i]>(803+offset0))
{
	if(input[i]>(805+offset0))
{
	if(input[i]==(807+offset0))
{
	vR[i] = 807;
}
else
{
	vR[i] = 806;
}

}
else
{
	if(input[i]==(805+offset0))
{
	vR[i] = 805;
}
else
{
	vR[i] = 804;
}

}

}
else
{
	if(input[i]>(801+offset0))
{
	if(input[i]==(803+offset0))
{
	vR[i] = 803;
}
else
{
	vR[i] = 802;
}

}
else
{
	if(input[i]==(801+offset0))
{
	vR[i] = 801;
}
else
{
	vR[i] = 800;
}

}

}

}

}

}
else
{
	if(input[i]>(783+offset0))
{
	if(input[i]>(791+offset0))
{
	if(input[i]>(795+offset0))
{
	if(input[i]>(797+offset0))
{
	if(input[i]==(799+offset0))
{
	vR[i] = 799;
}
else
{
	vR[i] = 798;
}

}
else
{
	if(input[i]==(797+offset0))
{
	vR[i] = 797;
}
else
{
	vR[i] = 796;
}

}

}
else
{
	if(input[i]>(793+offset0))
{
	if(input[i]==(795+offset0))
{
	vR[i] = 795;
}
else
{
	vR[i] = 794;
}

}
else
{
	if(input[i]==(793+offset0))
{
	vR[i] = 793;
}
else
{
	vR[i] = 792;
}

}

}

}
else
{
	if(input[i]>(787+offset0))
{
	if(input[i]>(789+offset0))
{
	if(input[i]==(791+offset0))
{
	vR[i] = 791;
}
else
{
	vR[i] = 790;
}

}
else
{
	if(input[i]==(789+offset0))
{
	vR[i] = 789;
}
else
{
	vR[i] = 788;
}

}

}
else
{
	if(input[i]>(785+offset0))
{
	if(input[i]==(787+offset0))
{
	vR[i] = 787;
}
else
{
	vR[i] = 786;
}

}
else
{
	if(input[i]==(785+offset0))
{
	vR[i] = 785;
}
else
{
	vR[i] = 784;
}

}

}

}

}
else
{
	if(input[i]>(775+offset0))
{
	if(input[i]>(779+offset0))
{
	if(input[i]>(781+offset0))
{
	if(input[i]==(783+offset0))
{
	vR[i] = 783;
}
else
{
	vR[i] = 782;
}

}
else
{
	if(input[i]==(781+offset0))
{
	vR[i] = 781;
}
else
{
	vR[i] = 780;
}

}

}
else
{
	if(input[i]>(777+offset0))
{
	if(input[i]==(779+offset0))
{
	vR[i] = 779;
}
else
{
	vR[i] = 778;
}

}
else
{
	if(input[i]==(777+offset0))
{
	vR[i] = 777;
}
else
{
	vR[i] = 776;
}

}

}

}
else
{
	if(input[i]>(771+offset0))
{
	if(input[i]>(773+offset0))
{
	if(input[i]==(775+offset0))
{
	vR[i] = 775;
}
else
{
	vR[i] = 774;
}

}
else
{
	if(input[i]==(773+offset0))
{
	vR[i] = 773;
}
else
{
	vR[i] = 772;
}

}

}
else
{
	if(input[i]>(769+offset0))
{
	if(input[i]==(771+offset0))
{
	vR[i] = 771;
}
else
{
	vR[i] = 770;
}

}
else
{
	if(input[i]==(769+offset0))
{
	vR[i] = 769;
}
else
{
	vR[i] = 768;
}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(639+offset0))
{
	if(input[i]>(703+offset0))
{
	if(input[i]>(735+offset0))
{
	if(input[i]>(751+offset0))
{
	if(input[i]>(759+offset0))
{
	if(input[i]>(763+offset0))
{
	if(input[i]>(765+offset0))
{
	if(input[i]==(767+offset0))
{
	vR[i] = 767;
}
else
{
	vR[i] = 766;
}

}
else
{
	if(input[i]==(765+offset0))
{
	vR[i] = 765;
}
else
{
	vR[i] = 764;
}

}

}
else
{
	if(input[i]>(761+offset0))
{
	if(input[i]==(763+offset0))
{
	vR[i] = 763;
}
else
{
	vR[i] = 762;
}

}
else
{
	if(input[i]==(761+offset0))
{
	vR[i] = 761;
}
else
{
	vR[i] = 760;
}

}

}

}
else
{
	if(input[i]>(755+offset0))
{
	if(input[i]>(757+offset0))
{
	if(input[i]==(759+offset0))
{
	vR[i] = 759;
}
else
{
	vR[i] = 758;
}

}
else
{
	if(input[i]==(757+offset0))
{
	vR[i] = 757;
}
else
{
	vR[i] = 756;
}

}

}
else
{
	if(input[i]>(753+offset0))
{
	if(input[i]==(755+offset0))
{
	vR[i] = 755;
}
else
{
	vR[i] = 754;
}

}
else
{
	if(input[i]==(753+offset0))
{
	vR[i] = 753;
}
else
{
	vR[i] = 752;
}

}

}

}

}
else
{
	if(input[i]>(743+offset0))
{
	if(input[i]>(747+offset0))
{
	if(input[i]>(749+offset0))
{
	if(input[i]==(751+offset0))
{
	vR[i] = 751;
}
else
{
	vR[i] = 750;
}

}
else
{
	if(input[i]==(749+offset0))
{
	vR[i] = 749;
}
else
{
	vR[i] = 748;
}

}

}
else
{
	if(input[i]>(745+offset0))
{
	if(input[i]==(747+offset0))
{
	vR[i] = 747;
}
else
{
	vR[i] = 746;
}

}
else
{
	if(input[i]==(745+offset0))
{
	vR[i] = 745;
}
else
{
	vR[i] = 744;
}

}

}

}
else
{
	if(input[i]>(739+offset0))
{
	if(input[i]>(741+offset0))
{
	if(input[i]==(743+offset0))
{
	vR[i] = 743;
}
else
{
	vR[i] = 742;
}

}
else
{
	if(input[i]==(741+offset0))
{
	vR[i] = 741;
}
else
{
	vR[i] = 740;
}

}

}
else
{
	if(input[i]>(737+offset0))
{
	if(input[i]==(739+offset0))
{
	vR[i] = 739;
}
else
{
	vR[i] = 738;
}

}
else
{
	if(input[i]==(737+offset0))
{
	vR[i] = 737;
}
else
{
	vR[i] = 736;
}

}

}

}

}

}
else
{
	if(input[i]>(719+offset0))
{
	if(input[i]>(727+offset0))
{
	if(input[i]>(731+offset0))
{
	if(input[i]>(733+offset0))
{
	if(input[i]==(735+offset0))
{
	vR[i] = 735;
}
else
{
	vR[i] = 734;
}

}
else
{
	if(input[i]==(733+offset0))
{
	vR[i] = 733;
}
else
{
	vR[i] = 732;
}

}

}
else
{
	if(input[i]>(729+offset0))
{
	if(input[i]==(731+offset0))
{
	vR[i] = 731;
}
else
{
	vR[i] = 730;
}

}
else
{
	if(input[i]==(729+offset0))
{
	vR[i] = 729;
}
else
{
	vR[i] = 728;
}

}

}

}
else
{
	if(input[i]>(723+offset0))
{
	if(input[i]>(725+offset0))
{
	if(input[i]==(727+offset0))
{
	vR[i] = 727;
}
else
{
	vR[i] = 726;
}

}
else
{
	if(input[i]==(725+offset0))
{
	vR[i] = 725;
}
else
{
	vR[i] = 724;
}

}

}
else
{
	if(input[i]>(721+offset0))
{
	if(input[i]==(723+offset0))
{
	vR[i] = 723;
}
else
{
	vR[i] = 722;
}

}
else
{
	if(input[i]==(721+offset0))
{
	vR[i] = 721;
}
else
{
	vR[i] = 720;
}

}

}

}

}
else
{
	if(input[i]>(711+offset0))
{
	if(input[i]>(715+offset0))
{
	if(input[i]>(717+offset0))
{
	if(input[i]==(719+offset0))
{
	vR[i] = 719;
}
else
{
	vR[i] = 718;
}

}
else
{
	if(input[i]==(717+offset0))
{
	vR[i] = 717;
}
else
{
	vR[i] = 716;
}

}

}
else
{
	if(input[i]>(713+offset0))
{
	if(input[i]==(715+offset0))
{
	vR[i] = 715;
}
else
{
	vR[i] = 714;
}

}
else
{
	if(input[i]==(713+offset0))
{
	vR[i] = 713;
}
else
{
	vR[i] = 712;
}

}

}

}
else
{
	if(input[i]>(707+offset0))
{
	if(input[i]>(709+offset0))
{
	if(input[i]==(711+offset0))
{
	vR[i] = 711;
}
else
{
	vR[i] = 710;
}

}
else
{
	if(input[i]==(709+offset0))
{
	vR[i] = 709;
}
else
{
	vR[i] = 708;
}

}

}
else
{
	if(input[i]>(705+offset0))
{
	if(input[i]==(707+offset0))
{
	vR[i] = 707;
}
else
{
	vR[i] = 706;
}

}
else
{
	if(input[i]==(705+offset0))
{
	vR[i] = 705;
}
else
{
	vR[i] = 704;
}

}

}

}

}

}

}
else
{
	if(input[i]>(671+offset0))
{
	if(input[i]>(687+offset0))
{
	if(input[i]>(695+offset0))
{
	if(input[i]>(699+offset0))
{
	if(input[i]>(701+offset0))
{
	if(input[i]==(703+offset0))
{
	vR[i] = 703;
}
else
{
	vR[i] = 702;
}

}
else
{
	if(input[i]==(701+offset0))
{
	vR[i] = 701;
}
else
{
	vR[i] = 700;
}

}

}
else
{
	if(input[i]>(697+offset0))
{
	if(input[i]==(699+offset0))
{
	vR[i] = 699;
}
else
{
	vR[i] = 698;
}

}
else
{
	if(input[i]==(697+offset0))
{
	vR[i] = 697;
}
else
{
	vR[i] = 696;
}

}

}

}
else
{
	if(input[i]>(691+offset0))
{
	if(input[i]>(693+offset0))
{
	if(input[i]==(695+offset0))
{
	vR[i] = 695;
}
else
{
	vR[i] = 694;
}

}
else
{
	if(input[i]==(693+offset0))
{
	vR[i] = 693;
}
else
{
	vR[i] = 692;
}

}

}
else
{
	if(input[i]>(689+offset0))
{
	if(input[i]==(691+offset0))
{
	vR[i] = 691;
}
else
{
	vR[i] = 690;
}

}
else
{
	if(input[i]==(689+offset0))
{
	vR[i] = 689;
}
else
{
	vR[i] = 688;
}

}

}

}

}
else
{
	if(input[i]>(679+offset0))
{
	if(input[i]>(683+offset0))
{
	if(input[i]>(685+offset0))
{
	if(input[i]==(687+offset0))
{
	vR[i] = 687;
}
else
{
	vR[i] = 686;
}

}
else
{
	if(input[i]==(685+offset0))
{
	vR[i] = 685;
}
else
{
	vR[i] = 684;
}

}

}
else
{
	if(input[i]>(681+offset0))
{
	if(input[i]==(683+offset0))
{
	vR[i] = 683;
}
else
{
	vR[i] = 682;
}

}
else
{
	if(input[i]==(681+offset0))
{
	vR[i] = 681;
}
else
{
	vR[i] = 680;
}

}

}

}
else
{
	if(input[i]>(675+offset0))
{
	if(input[i]>(677+offset0))
{
	if(input[i]==(679+offset0))
{
	vR[i] = 679;
}
else
{
	vR[i] = 678;
}

}
else
{
	if(input[i]==(677+offset0))
{
	vR[i] = 677;
}
else
{
	vR[i] = 676;
}

}

}
else
{
	if(input[i]>(673+offset0))
{
	if(input[i]==(675+offset0))
{
	vR[i] = 675;
}
else
{
	vR[i] = 674;
}

}
else
{
	if(input[i]==(673+offset0))
{
	vR[i] = 673;
}
else
{
	vR[i] = 672;
}

}

}

}

}

}
else
{
	if(input[i]>(655+offset0))
{
	if(input[i]>(663+offset0))
{
	if(input[i]>(667+offset0))
{
	if(input[i]>(669+offset0))
{
	if(input[i]==(671+offset0))
{
	vR[i] = 671;
}
else
{
	vR[i] = 670;
}

}
else
{
	if(input[i]==(669+offset0))
{
	vR[i] = 669;
}
else
{
	vR[i] = 668;
}

}

}
else
{
	if(input[i]>(665+offset0))
{
	if(input[i]==(667+offset0))
{
	vR[i] = 667;
}
else
{
	vR[i] = 666;
}

}
else
{
	if(input[i]==(665+offset0))
{
	vR[i] = 665;
}
else
{
	vR[i] = 664;
}

}

}

}
else
{
	if(input[i]>(659+offset0))
{
	if(input[i]>(661+offset0))
{
	if(input[i]==(663+offset0))
{
	vR[i] = 663;
}
else
{
	vR[i] = 662;
}

}
else
{
	if(input[i]==(661+offset0))
{
	vR[i] = 661;
}
else
{
	vR[i] = 660;
}

}

}
else
{
	if(input[i]>(657+offset0))
{
	if(input[i]==(659+offset0))
{
	vR[i] = 659;
}
else
{
	vR[i] = 658;
}

}
else
{
	if(input[i]==(657+offset0))
{
	vR[i] = 657;
}
else
{
	vR[i] = 656;
}

}

}

}

}
else
{
	if(input[i]>(647+offset0))
{
	if(input[i]>(651+offset0))
{
	if(input[i]>(653+offset0))
{
	if(input[i]==(655+offset0))
{
	vR[i] = 655;
}
else
{
	vR[i] = 654;
}

}
else
{
	if(input[i]==(653+offset0))
{
	vR[i] = 653;
}
else
{
	vR[i] = 652;
}

}

}
else
{
	if(input[i]>(649+offset0))
{
	if(input[i]==(651+offset0))
{
	vR[i] = 651;
}
else
{
	vR[i] = 650;
}

}
else
{
	if(input[i]==(649+offset0))
{
	vR[i] = 649;
}
else
{
	vR[i] = 648;
}

}

}

}
else
{
	if(input[i]>(643+offset0))
{
	if(input[i]>(645+offset0))
{
	if(input[i]==(647+offset0))
{
	vR[i] = 647;
}
else
{
	vR[i] = 646;
}

}
else
{
	if(input[i]==(645+offset0))
{
	vR[i] = 645;
}
else
{
	vR[i] = 644;
}

}

}
else
{
	if(input[i]>(641+offset0))
{
	if(input[i]==(643+offset0))
{
	vR[i] = 643;
}
else
{
	vR[i] = 642;
}

}
else
{
	if(input[i]==(641+offset0))
{
	vR[i] = 641;
}
else
{
	vR[i] = 640;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(575+offset0))
{
	if(input[i]>(607+offset0))
{
	if(input[i]>(623+offset0))
{
	if(input[i]>(631+offset0))
{
	if(input[i]>(635+offset0))
{
	if(input[i]>(637+offset0))
{
	if(input[i]==(639+offset0))
{
	vR[i] = 639;
}
else
{
	vR[i] = 638;
}

}
else
{
	if(input[i]==(637+offset0))
{
	vR[i] = 637;
}
else
{
	vR[i] = 636;
}

}

}
else
{
	if(input[i]>(633+offset0))
{
	if(input[i]==(635+offset0))
{
	vR[i] = 635;
}
else
{
	vR[i] = 634;
}

}
else
{
	if(input[i]==(633+offset0))
{
	vR[i] = 633;
}
else
{
	vR[i] = 632;
}

}

}

}
else
{
	if(input[i]>(627+offset0))
{
	if(input[i]>(629+offset0))
{
	if(input[i]==(631+offset0))
{
	vR[i] = 631;
}
else
{
	vR[i] = 630;
}

}
else
{
	if(input[i]==(629+offset0))
{
	vR[i] = 629;
}
else
{
	vR[i] = 628;
}

}

}
else
{
	if(input[i]>(625+offset0))
{
	if(input[i]==(627+offset0))
{
	vR[i] = 627;
}
else
{
	vR[i] = 626;
}

}
else
{
	if(input[i]==(625+offset0))
{
	vR[i] = 625;
}
else
{
	vR[i] = 624;
}

}

}

}

}
else
{
	if(input[i]>(615+offset0))
{
	if(input[i]>(619+offset0))
{
	if(input[i]>(621+offset0))
{
	if(input[i]==(623+offset0))
{
	vR[i] = 623;
}
else
{
	vR[i] = 622;
}

}
else
{
	if(input[i]==(621+offset0))
{
	vR[i] = 621;
}
else
{
	vR[i] = 620;
}

}

}
else
{
	if(input[i]>(617+offset0))
{
	if(input[i]==(619+offset0))
{
	vR[i] = 619;
}
else
{
	vR[i] = 618;
}

}
else
{
	if(input[i]==(617+offset0))
{
	vR[i] = 617;
}
else
{
	vR[i] = 616;
}

}

}

}
else
{
	if(input[i]>(611+offset0))
{
	if(input[i]>(613+offset0))
{
	if(input[i]==(615+offset0))
{
	vR[i] = 615;
}
else
{
	vR[i] = 614;
}

}
else
{
	if(input[i]==(613+offset0))
{
	vR[i] = 613;
}
else
{
	vR[i] = 612;
}

}

}
else
{
	if(input[i]>(609+offset0))
{
	if(input[i]==(611+offset0))
{
	vR[i] = 611;
}
else
{
	vR[i] = 610;
}

}
else
{
	if(input[i]==(609+offset0))
{
	vR[i] = 609;
}
else
{
	vR[i] = 608;
}

}

}

}

}

}
else
{
	if(input[i]>(591+offset0))
{
	if(input[i]>(599+offset0))
{
	if(input[i]>(603+offset0))
{
	if(input[i]>(605+offset0))
{
	if(input[i]==(607+offset0))
{
	vR[i] = 607;
}
else
{
	vR[i] = 606;
}

}
else
{
	if(input[i]==(605+offset0))
{
	vR[i] = 605;
}
else
{
	vR[i] = 604;
}

}

}
else
{
	if(input[i]>(601+offset0))
{
	if(input[i]==(603+offset0))
{
	vR[i] = 603;
}
else
{
	vR[i] = 602;
}

}
else
{
	if(input[i]==(601+offset0))
{
	vR[i] = 601;
}
else
{
	vR[i] = 600;
}

}

}

}
else
{
	if(input[i]>(595+offset0))
{
	if(input[i]>(597+offset0))
{
	if(input[i]==(599+offset0))
{
	vR[i] = 599;
}
else
{
	vR[i] = 598;
}

}
else
{
	if(input[i]==(597+offset0))
{
	vR[i] = 597;
}
else
{
	vR[i] = 596;
}

}

}
else
{
	if(input[i]>(593+offset0))
{
	if(input[i]==(595+offset0))
{
	vR[i] = 595;
}
else
{
	vR[i] = 594;
}

}
else
{
	if(input[i]==(593+offset0))
{
	vR[i] = 593;
}
else
{
	vR[i] = 592;
}

}

}

}

}
else
{
	if(input[i]>(583+offset0))
{
	if(input[i]>(587+offset0))
{
	if(input[i]>(589+offset0))
{
	if(input[i]==(591+offset0))
{
	vR[i] = 591;
}
else
{
	vR[i] = 590;
}

}
else
{
	if(input[i]==(589+offset0))
{
	vR[i] = 589;
}
else
{
	vR[i] = 588;
}

}

}
else
{
	if(input[i]>(585+offset0))
{
	if(input[i]==(587+offset0))
{
	vR[i] = 587;
}
else
{
	vR[i] = 586;
}

}
else
{
	if(input[i]==(585+offset0))
{
	vR[i] = 585;
}
else
{
	vR[i] = 584;
}

}

}

}
else
{
	if(input[i]>(579+offset0))
{
	if(input[i]>(581+offset0))
{
	if(input[i]==(583+offset0))
{
	vR[i] = 583;
}
else
{
	vR[i] = 582;
}

}
else
{
	if(input[i]==(581+offset0))
{
	vR[i] = 581;
}
else
{
	vR[i] = 580;
}

}

}
else
{
	if(input[i]>(577+offset0))
{
	if(input[i]==(579+offset0))
{
	vR[i] = 579;
}
else
{
	vR[i] = 578;
}

}
else
{
	if(input[i]==(577+offset0))
{
	vR[i] = 577;
}
else
{
	vR[i] = 576;
}

}

}

}

}

}

}
else
{
	if(input[i]>(543+offset0))
{
	if(input[i]>(559+offset0))
{
	if(input[i]>(567+offset0))
{
	if(input[i]>(571+offset0))
{
	if(input[i]>(573+offset0))
{
	if(input[i]==(575+offset0))
{
	vR[i] = 575;
}
else
{
	vR[i] = 574;
}

}
else
{
	if(input[i]==(573+offset0))
{
	vR[i] = 573;
}
else
{
	vR[i] = 572;
}

}

}
else
{
	if(input[i]>(569+offset0))
{
	if(input[i]==(571+offset0))
{
	vR[i] = 571;
}
else
{
	vR[i] = 570;
}

}
else
{
	if(input[i]==(569+offset0))
{
	vR[i] = 569;
}
else
{
	vR[i] = 568;
}

}

}

}
else
{
	if(input[i]>(563+offset0))
{
	if(input[i]>(565+offset0))
{
	if(input[i]==(567+offset0))
{
	vR[i] = 567;
}
else
{
	vR[i] = 566;
}

}
else
{
	if(input[i]==(565+offset0))
{
	vR[i] = 565;
}
else
{
	vR[i] = 564;
}

}

}
else
{
	if(input[i]>(561+offset0))
{
	if(input[i]==(563+offset0))
{
	vR[i] = 563;
}
else
{
	vR[i] = 562;
}

}
else
{
	if(input[i]==(561+offset0))
{
	vR[i] = 561;
}
else
{
	vR[i] = 560;
}

}

}

}

}
else
{
	if(input[i]>(551+offset0))
{
	if(input[i]>(555+offset0))
{
	if(input[i]>(557+offset0))
{
	if(input[i]==(559+offset0))
{
	vR[i] = 559;
}
else
{
	vR[i] = 558;
}

}
else
{
	if(input[i]==(557+offset0))
{
	vR[i] = 557;
}
else
{
	vR[i] = 556;
}

}

}
else
{
	if(input[i]>(553+offset0))
{
	if(input[i]==(555+offset0))
{
	vR[i] = 555;
}
else
{
	vR[i] = 554;
}

}
else
{
	if(input[i]==(553+offset0))
{
	vR[i] = 553;
}
else
{
	vR[i] = 552;
}

}

}

}
else
{
	if(input[i]>(547+offset0))
{
	if(input[i]>(549+offset0))
{
	if(input[i]==(551+offset0))
{
	vR[i] = 551;
}
else
{
	vR[i] = 550;
}

}
else
{
	if(input[i]==(549+offset0))
{
	vR[i] = 549;
}
else
{
	vR[i] = 548;
}

}

}
else
{
	if(input[i]>(545+offset0))
{
	if(input[i]==(547+offset0))
{
	vR[i] = 547;
}
else
{
	vR[i] = 546;
}

}
else
{
	if(input[i]==(545+offset0))
{
	vR[i] = 545;
}
else
{
	vR[i] = 544;
}

}

}

}

}

}
else
{
	if(input[i]>(527+offset0))
{
	if(input[i]>(535+offset0))
{
	if(input[i]>(539+offset0))
{
	if(input[i]>(541+offset0))
{
	if(input[i]==(543+offset0))
{
	vR[i] = 543;
}
else
{
	vR[i] = 542;
}

}
else
{
	if(input[i]==(541+offset0))
{
	vR[i] = 541;
}
else
{
	vR[i] = 540;
}

}

}
else
{
	if(input[i]>(537+offset0))
{
	if(input[i]==(539+offset0))
{
	vR[i] = 539;
}
else
{
	vR[i] = 538;
}

}
else
{
	if(input[i]==(537+offset0))
{
	vR[i] = 537;
}
else
{
	vR[i] = 536;
}

}

}

}
else
{
	if(input[i]>(531+offset0))
{
	if(input[i]>(533+offset0))
{
	if(input[i]==(535+offset0))
{
	vR[i] = 535;
}
else
{
	vR[i] = 534;
}

}
else
{
	if(input[i]==(533+offset0))
{
	vR[i] = 533;
}
else
{
	vR[i] = 532;
}

}

}
else
{
	if(input[i]>(529+offset0))
{
	if(input[i]==(531+offset0))
{
	vR[i] = 531;
}
else
{
	vR[i] = 530;
}

}
else
{
	if(input[i]==(529+offset0))
{
	vR[i] = 529;
}
else
{
	vR[i] = 528;
}

}

}

}

}
else
{
	if(input[i]>(519+offset0))
{
	if(input[i]>(523+offset0))
{
	if(input[i]>(525+offset0))
{
	if(input[i]==(527+offset0))
{
	vR[i] = 527;
}
else
{
	vR[i] = 526;
}

}
else
{
	if(input[i]==(525+offset0))
{
	vR[i] = 525;
}
else
{
	vR[i] = 524;
}

}

}
else
{
	if(input[i]>(521+offset0))
{
	if(input[i]==(523+offset0))
{
	vR[i] = 523;
}
else
{
	vR[i] = 522;
}

}
else
{
	if(input[i]==(521+offset0))
{
	vR[i] = 521;
}
else
{
	vR[i] = 520;
}

}

}

}
else
{
	if(input[i]>(515+offset0))
{
	if(input[i]>(517+offset0))
{
	if(input[i]==(519+offset0))
{
	vR[i] = 519;
}
else
{
	vR[i] = 518;
}

}
else
{
	if(input[i]==(517+offset0))
{
	vR[i] = 517;
}
else
{
	vR[i] = 516;
}

}

}
else
{
	if(input[i]>(513+offset0))
{
	if(input[i]==(515+offset0))
{
	vR[i] = 515;
}
else
{
	vR[i] = 514;
}

}
else
{
	if(input[i]==(513+offset0))
{
	vR[i] = 513;
}
else
{
	vR[i] = 512;
}

}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(255+offset0))
{
	if(input[i]>(383+offset0))
{
	if(input[i]>(447+offset0))
{
	if(input[i]>(479+offset0))
{
	if(input[i]>(495+offset0))
{
	if(input[i]>(503+offset0))
{
	if(input[i]>(507+offset0))
{
	if(input[i]>(509+offset0))
{
	if(input[i]==(511+offset0))
{
	vR[i] = 511;
}
else
{
	vR[i] = 510;
}

}
else
{
	if(input[i]==(509+offset0))
{
	vR[i] = 509;
}
else
{
	vR[i] = 508;
}

}

}
else
{
	if(input[i]>(505+offset0))
{
	if(input[i]==(507+offset0))
{
	vR[i] = 507;
}
else
{
	vR[i] = 506;
}

}
else
{
	if(input[i]==(505+offset0))
{
	vR[i] = 505;
}
else
{
	vR[i] = 504;
}

}

}

}
else
{
	if(input[i]>(499+offset0))
{
	if(input[i]>(501+offset0))
{
	if(input[i]==(503+offset0))
{
	vR[i] = 503;
}
else
{
	vR[i] = 502;
}

}
else
{
	if(input[i]==(501+offset0))
{
	vR[i] = 501;
}
else
{
	vR[i] = 500;
}

}

}
else
{
	if(input[i]>(497+offset0))
{
	if(input[i]==(499+offset0))
{
	vR[i] = 499;
}
else
{
	vR[i] = 498;
}

}
else
{
	if(input[i]==(497+offset0))
{
	vR[i] = 497;
}
else
{
	vR[i] = 496;
}

}

}

}

}
else
{
	if(input[i]>(487+offset0))
{
	if(input[i]>(491+offset0))
{
	if(input[i]>(493+offset0))
{
	if(input[i]==(495+offset0))
{
	vR[i] = 495;
}
else
{
	vR[i] = 494;
}

}
else
{
	if(input[i]==(493+offset0))
{
	vR[i] = 493;
}
else
{
	vR[i] = 492;
}

}

}
else
{
	if(input[i]>(489+offset0))
{
	if(input[i]==(491+offset0))
{
	vR[i] = 491;
}
else
{
	vR[i] = 490;
}

}
else
{
	if(input[i]==(489+offset0))
{
	vR[i] = 489;
}
else
{
	vR[i] = 488;
}

}

}

}
else
{
	if(input[i]>(483+offset0))
{
	if(input[i]>(485+offset0))
{
	if(input[i]==(487+offset0))
{
	vR[i] = 487;
}
else
{
	vR[i] = 486;
}

}
else
{
	if(input[i]==(485+offset0))
{
	vR[i] = 485;
}
else
{
	vR[i] = 484;
}

}

}
else
{
	if(input[i]>(481+offset0))
{
	if(input[i]==(483+offset0))
{
	vR[i] = 483;
}
else
{
	vR[i] = 482;
}

}
else
{
	if(input[i]==(481+offset0))
{
	vR[i] = 481;
}
else
{
	vR[i] = 480;
}

}

}

}

}

}
else
{
	if(input[i]>(463+offset0))
{
	if(input[i]>(471+offset0))
{
	if(input[i]>(475+offset0))
{
	if(input[i]>(477+offset0))
{
	if(input[i]==(479+offset0))
{
	vR[i] = 479;
}
else
{
	vR[i] = 478;
}

}
else
{
	if(input[i]==(477+offset0))
{
	vR[i] = 477;
}
else
{
	vR[i] = 476;
}

}

}
else
{
	if(input[i]>(473+offset0))
{
	if(input[i]==(475+offset0))
{
	vR[i] = 475;
}
else
{
	vR[i] = 474;
}

}
else
{
	if(input[i]==(473+offset0))
{
	vR[i] = 473;
}
else
{
	vR[i] = 472;
}

}

}

}
else
{
	if(input[i]>(467+offset0))
{
	if(input[i]>(469+offset0))
{
	if(input[i]==(471+offset0))
{
	vR[i] = 471;
}
else
{
	vR[i] = 470;
}

}
else
{
	if(input[i]==(469+offset0))
{
	vR[i] = 469;
}
else
{
	vR[i] = 468;
}

}

}
else
{
	if(input[i]>(465+offset0))
{
	if(input[i]==(467+offset0))
{
	vR[i] = 467;
}
else
{
	vR[i] = 466;
}

}
else
{
	if(input[i]==(465+offset0))
{
	vR[i] = 465;
}
else
{
	vR[i] = 464;
}

}

}

}

}
else
{
	if(input[i]>(455+offset0))
{
	if(input[i]>(459+offset0))
{
	if(input[i]>(461+offset0))
{
	if(input[i]==(463+offset0))
{
	vR[i] = 463;
}
else
{
	vR[i] = 462;
}

}
else
{
	if(input[i]==(461+offset0))
{
	vR[i] = 461;
}
else
{
	vR[i] = 460;
}

}

}
else
{
	if(input[i]>(457+offset0))
{
	if(input[i]==(459+offset0))
{
	vR[i] = 459;
}
else
{
	vR[i] = 458;
}

}
else
{
	if(input[i]==(457+offset0))
{
	vR[i] = 457;
}
else
{
	vR[i] = 456;
}

}

}

}
else
{
	if(input[i]>(451+offset0))
{
	if(input[i]>(453+offset0))
{
	if(input[i]==(455+offset0))
{
	vR[i] = 455;
}
else
{
	vR[i] = 454;
}

}
else
{
	if(input[i]==(453+offset0))
{
	vR[i] = 453;
}
else
{
	vR[i] = 452;
}

}

}
else
{
	if(input[i]>(449+offset0))
{
	if(input[i]==(451+offset0))
{
	vR[i] = 451;
}
else
{
	vR[i] = 450;
}

}
else
{
	if(input[i]==(449+offset0))
{
	vR[i] = 449;
}
else
{
	vR[i] = 448;
}

}

}

}

}

}

}
else
{
	if(input[i]>(415+offset0))
{
	if(input[i]>(431+offset0))
{
	if(input[i]>(439+offset0))
{
	if(input[i]>(443+offset0))
{
	if(input[i]>(445+offset0))
{
	if(input[i]==(447+offset0))
{
	vR[i] = 447;
}
else
{
	vR[i] = 446;
}

}
else
{
	if(input[i]==(445+offset0))
{
	vR[i] = 445;
}
else
{
	vR[i] = 444;
}

}

}
else
{
	if(input[i]>(441+offset0))
{
	if(input[i]==(443+offset0))
{
	vR[i] = 443;
}
else
{
	vR[i] = 442;
}

}
else
{
	if(input[i]==(441+offset0))
{
	vR[i] = 441;
}
else
{
	vR[i] = 440;
}

}

}

}
else
{
	if(input[i]>(435+offset0))
{
	if(input[i]>(437+offset0))
{
	if(input[i]==(439+offset0))
{
	vR[i] = 439;
}
else
{
	vR[i] = 438;
}

}
else
{
	if(input[i]==(437+offset0))
{
	vR[i] = 437;
}
else
{
	vR[i] = 436;
}

}

}
else
{
	if(input[i]>(433+offset0))
{
	if(input[i]==(435+offset0))
{
	vR[i] = 435;
}
else
{
	vR[i] = 434;
}

}
else
{
	if(input[i]==(433+offset0))
{
	vR[i] = 433;
}
else
{
	vR[i] = 432;
}

}

}

}

}
else
{
	if(input[i]>(423+offset0))
{
	if(input[i]>(427+offset0))
{
	if(input[i]>(429+offset0))
{
	if(input[i]==(431+offset0))
{
	vR[i] = 431;
}
else
{
	vR[i] = 430;
}

}
else
{
	if(input[i]==(429+offset0))
{
	vR[i] = 429;
}
else
{
	vR[i] = 428;
}

}

}
else
{
	if(input[i]>(425+offset0))
{
	if(input[i]==(427+offset0))
{
	vR[i] = 427;
}
else
{
	vR[i] = 426;
}

}
else
{
	if(input[i]==(425+offset0))
{
	vR[i] = 425;
}
else
{
	vR[i] = 424;
}

}

}

}
else
{
	if(input[i]>(419+offset0))
{
	if(input[i]>(421+offset0))
{
	if(input[i]==(423+offset0))
{
	vR[i] = 423;
}
else
{
	vR[i] = 422;
}

}
else
{
	if(input[i]==(421+offset0))
{
	vR[i] = 421;
}
else
{
	vR[i] = 420;
}

}

}
else
{
	if(input[i]>(417+offset0))
{
	if(input[i]==(419+offset0))
{
	vR[i] = 419;
}
else
{
	vR[i] = 418;
}

}
else
{
	if(input[i]==(417+offset0))
{
	vR[i] = 417;
}
else
{
	vR[i] = 416;
}

}

}

}

}

}
else
{
	if(input[i]>(399+offset0))
{
	if(input[i]>(407+offset0))
{
	if(input[i]>(411+offset0))
{
	if(input[i]>(413+offset0))
{
	if(input[i]==(415+offset0))
{
	vR[i] = 415;
}
else
{
	vR[i] = 414;
}

}
else
{
	if(input[i]==(413+offset0))
{
	vR[i] = 413;
}
else
{
	vR[i] = 412;
}

}

}
else
{
	if(input[i]>(409+offset0))
{
	if(input[i]==(411+offset0))
{
	vR[i] = 411;
}
else
{
	vR[i] = 410;
}

}
else
{
	if(input[i]==(409+offset0))
{
	vR[i] = 409;
}
else
{
	vR[i] = 408;
}

}

}

}
else
{
	if(input[i]>(403+offset0))
{
	if(input[i]>(405+offset0))
{
	if(input[i]==(407+offset0))
{
	vR[i] = 407;
}
else
{
	vR[i] = 406;
}

}
else
{
	if(input[i]==(405+offset0))
{
	vR[i] = 405;
}
else
{
	vR[i] = 404;
}

}

}
else
{
	if(input[i]>(401+offset0))
{
	if(input[i]==(403+offset0))
{
	vR[i] = 403;
}
else
{
	vR[i] = 402;
}

}
else
{
	if(input[i]==(401+offset0))
{
	vR[i] = 401;
}
else
{
	vR[i] = 400;
}

}

}

}

}
else
{
	if(input[i]>(391+offset0))
{
	if(input[i]>(395+offset0))
{
	if(input[i]>(397+offset0))
{
	if(input[i]==(399+offset0))
{
	vR[i] = 399;
}
else
{
	vR[i] = 398;
}

}
else
{
	if(input[i]==(397+offset0))
{
	vR[i] = 397;
}
else
{
	vR[i] = 396;
}

}

}
else
{
	if(input[i]>(393+offset0))
{
	if(input[i]==(395+offset0))
{
	vR[i] = 395;
}
else
{
	vR[i] = 394;
}

}
else
{
	if(input[i]==(393+offset0))
{
	vR[i] = 393;
}
else
{
	vR[i] = 392;
}

}

}

}
else
{
	if(input[i]>(387+offset0))
{
	if(input[i]>(389+offset0))
{
	if(input[i]==(391+offset0))
{
	vR[i] = 391;
}
else
{
	vR[i] = 390;
}

}
else
{
	if(input[i]==(389+offset0))
{
	vR[i] = 389;
}
else
{
	vR[i] = 388;
}

}

}
else
{
	if(input[i]>(385+offset0))
{
	if(input[i]==(387+offset0))
{
	vR[i] = 387;
}
else
{
	vR[i] = 386;
}

}
else
{
	if(input[i]==(385+offset0))
{
	vR[i] = 385;
}
else
{
	vR[i] = 384;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(319+offset0))
{
	if(input[i]>(351+offset0))
{
	if(input[i]>(367+offset0))
{
	if(input[i]>(375+offset0))
{
	if(input[i]>(379+offset0))
{
	if(input[i]>(381+offset0))
{
	if(input[i]==(383+offset0))
{
	vR[i] = 383;
}
else
{
	vR[i] = 382;
}

}
else
{
	if(input[i]==(381+offset0))
{
	vR[i] = 381;
}
else
{
	vR[i] = 380;
}

}

}
else
{
	if(input[i]>(377+offset0))
{
	if(input[i]==(379+offset0))
{
	vR[i] = 379;
}
else
{
	vR[i] = 378;
}

}
else
{
	if(input[i]==(377+offset0))
{
	vR[i] = 377;
}
else
{
	vR[i] = 376;
}

}

}

}
else
{
	if(input[i]>(371+offset0))
{
	if(input[i]>(373+offset0))
{
	if(input[i]==(375+offset0))
{
	vR[i] = 375;
}
else
{
	vR[i] = 374;
}

}
else
{
	if(input[i]==(373+offset0))
{
	vR[i] = 373;
}
else
{
	vR[i] = 372;
}

}

}
else
{
	if(input[i]>(369+offset0))
{
	if(input[i]==(371+offset0))
{
	vR[i] = 371;
}
else
{
	vR[i] = 370;
}

}
else
{
	if(input[i]==(369+offset0))
{
	vR[i] = 369;
}
else
{
	vR[i] = 368;
}

}

}

}

}
else
{
	if(input[i]>(359+offset0))
{
	if(input[i]>(363+offset0))
{
	if(input[i]>(365+offset0))
{
	if(input[i]==(367+offset0))
{
	vR[i] = 367;
}
else
{
	vR[i] = 366;
}

}
else
{
	if(input[i]==(365+offset0))
{
	vR[i] = 365;
}
else
{
	vR[i] = 364;
}

}

}
else
{
	if(input[i]>(361+offset0))
{
	if(input[i]==(363+offset0))
{
	vR[i] = 363;
}
else
{
	vR[i] = 362;
}

}
else
{
	if(input[i]==(361+offset0))
{
	vR[i] = 361;
}
else
{
	vR[i] = 360;
}

}

}

}
else
{
	if(input[i]>(355+offset0))
{
	if(input[i]>(357+offset0))
{
	if(input[i]==(359+offset0))
{
	vR[i] = 359;
}
else
{
	vR[i] = 358;
}

}
else
{
	if(input[i]==(357+offset0))
{
	vR[i] = 357;
}
else
{
	vR[i] = 356;
}

}

}
else
{
	if(input[i]>(353+offset0))
{
	if(input[i]==(355+offset0))
{
	vR[i] = 355;
}
else
{
	vR[i] = 354;
}

}
else
{
	if(input[i]==(353+offset0))
{
	vR[i] = 353;
}
else
{
	vR[i] = 352;
}

}

}

}

}

}
else
{
	if(input[i]>(335+offset0))
{
	if(input[i]>(343+offset0))
{
	if(input[i]>(347+offset0))
{
	if(input[i]>(349+offset0))
{
	if(input[i]==(351+offset0))
{
	vR[i] = 351;
}
else
{
	vR[i] = 350;
}

}
else
{
	if(input[i]==(349+offset0))
{
	vR[i] = 349;
}
else
{
	vR[i] = 348;
}

}

}
else
{
	if(input[i]>(345+offset0))
{
	if(input[i]==(347+offset0))
{
	vR[i] = 347;
}
else
{
	vR[i] = 346;
}

}
else
{
	if(input[i]==(345+offset0))
{
	vR[i] = 345;
}
else
{
	vR[i] = 344;
}

}

}

}
else
{
	if(input[i]>(339+offset0))
{
	if(input[i]>(341+offset0))
{
	if(input[i]==(343+offset0))
{
	vR[i] = 343;
}
else
{
	vR[i] = 342;
}

}
else
{
	if(input[i]==(341+offset0))
{
	vR[i] = 341;
}
else
{
	vR[i] = 340;
}

}

}
else
{
	if(input[i]>(337+offset0))
{
	if(input[i]==(339+offset0))
{
	vR[i] = 339;
}
else
{
	vR[i] = 338;
}

}
else
{
	if(input[i]==(337+offset0))
{
	vR[i] = 337;
}
else
{
	vR[i] = 336;
}

}

}

}

}
else
{
	if(input[i]>(327+offset0))
{
	if(input[i]>(331+offset0))
{
	if(input[i]>(333+offset0))
{
	if(input[i]==(335+offset0))
{
	vR[i] = 335;
}
else
{
	vR[i] = 334;
}

}
else
{
	if(input[i]==(333+offset0))
{
	vR[i] = 333;
}
else
{
	vR[i] = 332;
}

}

}
else
{
	if(input[i]>(329+offset0))
{
	if(input[i]==(331+offset0))
{
	vR[i] = 331;
}
else
{
	vR[i] = 330;
}

}
else
{
	if(input[i]==(329+offset0))
{
	vR[i] = 329;
}
else
{
	vR[i] = 328;
}

}

}

}
else
{
	if(input[i]>(323+offset0))
{
	if(input[i]>(325+offset0))
{
	if(input[i]==(327+offset0))
{
	vR[i] = 327;
}
else
{
	vR[i] = 326;
}

}
else
{
	if(input[i]==(325+offset0))
{
	vR[i] = 325;
}
else
{
	vR[i] = 324;
}

}

}
else
{
	if(input[i]>(321+offset0))
{
	if(input[i]==(323+offset0))
{
	vR[i] = 323;
}
else
{
	vR[i] = 322;
}

}
else
{
	if(input[i]==(321+offset0))
{
	vR[i] = 321;
}
else
{
	vR[i] = 320;
}

}

}

}

}

}

}
else
{
	if(input[i]>(287+offset0))
{
	if(input[i]>(303+offset0))
{
	if(input[i]>(311+offset0))
{
	if(input[i]>(315+offset0))
{
	if(input[i]>(317+offset0))
{
	if(input[i]==(319+offset0))
{
	vR[i] = 319;
}
else
{
	vR[i] = 318;
}

}
else
{
	if(input[i]==(317+offset0))
{
	vR[i] = 317;
}
else
{
	vR[i] = 316;
}

}

}
else
{
	if(input[i]>(313+offset0))
{
	if(input[i]==(315+offset0))
{
	vR[i] = 315;
}
else
{
	vR[i] = 314;
}

}
else
{
	if(input[i]==(313+offset0))
{
	vR[i] = 313;
}
else
{
	vR[i] = 312;
}

}

}

}
else
{
	if(input[i]>(307+offset0))
{
	if(input[i]>(309+offset0))
{
	if(input[i]==(311+offset0))
{
	vR[i] = 311;
}
else
{
	vR[i] = 310;
}

}
else
{
	if(input[i]==(309+offset0))
{
	vR[i] = 309;
}
else
{
	vR[i] = 308;
}

}

}
else
{
	if(input[i]>(305+offset0))
{
	if(input[i]==(307+offset0))
{
	vR[i] = 307;
}
else
{
	vR[i] = 306;
}

}
else
{
	if(input[i]==(305+offset0))
{
	vR[i] = 305;
}
else
{
	vR[i] = 304;
}

}

}

}

}
else
{
	if(input[i]>(295+offset0))
{
	if(input[i]>(299+offset0))
{
	if(input[i]>(301+offset0))
{
	if(input[i]==(303+offset0))
{
	vR[i] = 303;
}
else
{
	vR[i] = 302;
}

}
else
{
	if(input[i]==(301+offset0))
{
	vR[i] = 301;
}
else
{
	vR[i] = 300;
}

}

}
else
{
	if(input[i]>(297+offset0))
{
	if(input[i]==(299+offset0))
{
	vR[i] = 299;
}
else
{
	vR[i] = 298;
}

}
else
{
	if(input[i]==(297+offset0))
{
	vR[i] = 297;
}
else
{
	vR[i] = 296;
}

}

}

}
else
{
	if(input[i]>(291+offset0))
{
	if(input[i]>(293+offset0))
{
	if(input[i]==(295+offset0))
{
	vR[i] = 295;
}
else
{
	vR[i] = 294;
}

}
else
{
	if(input[i]==(293+offset0))
{
	vR[i] = 293;
}
else
{
	vR[i] = 292;
}

}

}
else
{
	if(input[i]>(289+offset0))
{
	if(input[i]==(291+offset0))
{
	vR[i] = 291;
}
else
{
	vR[i] = 290;
}

}
else
{
	if(input[i]==(289+offset0))
{
	vR[i] = 289;
}
else
{
	vR[i] = 288;
}

}

}

}

}

}
else
{
	if(input[i]>(271+offset0))
{
	if(input[i]>(279+offset0))
{
	if(input[i]>(283+offset0))
{
	if(input[i]>(285+offset0))
{
	if(input[i]==(287+offset0))
{
	vR[i] = 287;
}
else
{
	vR[i] = 286;
}

}
else
{
	if(input[i]==(285+offset0))
{
	vR[i] = 285;
}
else
{
	vR[i] = 284;
}

}

}
else
{
	if(input[i]>(281+offset0))
{
	if(input[i]==(283+offset0))
{
	vR[i] = 283;
}
else
{
	vR[i] = 282;
}

}
else
{
	if(input[i]==(281+offset0))
{
	vR[i] = 281;
}
else
{
	vR[i] = 280;
}

}

}

}
else
{
	if(input[i]>(275+offset0))
{
	if(input[i]>(277+offset0))
{
	if(input[i]==(279+offset0))
{
	vR[i] = 279;
}
else
{
	vR[i] = 278;
}

}
else
{
	if(input[i]==(277+offset0))
{
	vR[i] = 277;
}
else
{
	vR[i] = 276;
}

}

}
else
{
	if(input[i]>(273+offset0))
{
	if(input[i]==(275+offset0))
{
	vR[i] = 275;
}
else
{
	vR[i] = 274;
}

}
else
{
	if(input[i]==(273+offset0))
{
	vR[i] = 273;
}
else
{
	vR[i] = 272;
}

}

}

}

}
else
{
	if(input[i]>(263+offset0))
{
	if(input[i]>(267+offset0))
{
	if(input[i]>(269+offset0))
{
	if(input[i]==(271+offset0))
{
	vR[i] = 271;
}
else
{
	vR[i] = 270;
}

}
else
{
	if(input[i]==(269+offset0))
{
	vR[i] = 269;
}
else
{
	vR[i] = 268;
}

}

}
else
{
	if(input[i]>(265+offset0))
{
	if(input[i]==(267+offset0))
{
	vR[i] = 267;
}
else
{
	vR[i] = 266;
}

}
else
{
	if(input[i]==(265+offset0))
{
	vR[i] = 265;
}
else
{
	vR[i] = 264;
}

}

}

}
else
{
	if(input[i]>(259+offset0))
{
	if(input[i]>(261+offset0))
{
	if(input[i]==(263+offset0))
{
	vR[i] = 263;
}
else
{
	vR[i] = 262;
}

}
else
{
	if(input[i]==(261+offset0))
{
	vR[i] = 261;
}
else
{
	vR[i] = 260;
}

}

}
else
{
	if(input[i]>(257+offset0))
{
	if(input[i]==(259+offset0))
{
	vR[i] = 259;
}
else
{
	vR[i] = 258;
}

}
else
{
	if(input[i]==(257+offset0))
{
	vR[i] = 257;
}
else
{
	vR[i] = 256;
}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(127+offset0))
{
	if(input[i]>(191+offset0))
{
	if(input[i]>(223+offset0))
{
	if(input[i]>(239+offset0))
{
	if(input[i]>(247+offset0))
{
	if(input[i]>(251+offset0))
{
	if(input[i]>(253+offset0))
{
	if(input[i]==(255+offset0))
{
	vR[i] = 255;
}
else
{
	vR[i] = 254;
}

}
else
{
	if(input[i]==(253+offset0))
{
	vR[i] = 253;
}
else
{
	vR[i] = 252;
}

}

}
else
{
	if(input[i]>(249+offset0))
{
	if(input[i]==(251+offset0))
{
	vR[i] = 251;
}
else
{
	vR[i] = 250;
}

}
else
{
	if(input[i]==(249+offset0))
{
	vR[i] = 249;
}
else
{
	vR[i] = 248;
}

}

}

}
else
{
	if(input[i]>(243+offset0))
{
	if(input[i]>(245+offset0))
{
	if(input[i]==(247+offset0))
{
	vR[i] = 247;
}
else
{
	vR[i] = 246;
}

}
else
{
	if(input[i]==(245+offset0))
{
	vR[i] = 245;
}
else
{
	vR[i] = 244;
}

}

}
else
{
	if(input[i]>(241+offset0))
{
	if(input[i]==(243+offset0))
{
	vR[i] = 243;
}
else
{
	vR[i] = 242;
}

}
else
{
	if(input[i]==(241+offset0))
{
	vR[i] = 241;
}
else
{
	vR[i] = 240;
}

}

}

}

}
else
{
	if(input[i]>(231+offset0))
{
	if(input[i]>(235+offset0))
{
	if(input[i]>(237+offset0))
{
	if(input[i]==(239+offset0))
{
	vR[i] = 239;
}
else
{
	vR[i] = 238;
}

}
else
{
	if(input[i]==(237+offset0))
{
	vR[i] = 237;
}
else
{
	vR[i] = 236;
}

}

}
else
{
	if(input[i]>(233+offset0))
{
	if(input[i]==(235+offset0))
{
	vR[i] = 235;
}
else
{
	vR[i] = 234;
}

}
else
{
	if(input[i]==(233+offset0))
{
	vR[i] = 233;
}
else
{
	vR[i] = 232;
}

}

}

}
else
{
	if(input[i]>(227+offset0))
{
	if(input[i]>(229+offset0))
{
	if(input[i]==(231+offset0))
{
	vR[i] = 231;
}
else
{
	vR[i] = 230;
}

}
else
{
	if(input[i]==(229+offset0))
{
	vR[i] = 229;
}
else
{
	vR[i] = 228;
}

}

}
else
{
	if(input[i]>(225+offset0))
{
	if(input[i]==(227+offset0))
{
	vR[i] = 227;
}
else
{
	vR[i] = 226;
}

}
else
{
	if(input[i]==(225+offset0))
{
	vR[i] = 225;
}
else
{
	vR[i] = 224;
}

}

}

}

}

}
else
{
	if(input[i]>(207+offset0))
{
	if(input[i]>(215+offset0))
{
	if(input[i]>(219+offset0))
{
	if(input[i]>(221+offset0))
{
	if(input[i]==(223+offset0))
{
	vR[i] = 223;
}
else
{
	vR[i] = 222;
}

}
else
{
	if(input[i]==(221+offset0))
{
	vR[i] = 221;
}
else
{
	vR[i] = 220;
}

}

}
else
{
	if(input[i]>(217+offset0))
{
	if(input[i]==(219+offset0))
{
	vR[i] = 219;
}
else
{
	vR[i] = 218;
}

}
else
{
	if(input[i]==(217+offset0))
{
	vR[i] = 217;
}
else
{
	vR[i] = 216;
}

}

}

}
else
{
	if(input[i]>(211+offset0))
{
	if(input[i]>(213+offset0))
{
	if(input[i]==(215+offset0))
{
	vR[i] = 215;
}
else
{
	vR[i] = 214;
}

}
else
{
	if(input[i]==(213+offset0))
{
	vR[i] = 213;
}
else
{
	vR[i] = 212;
}

}

}
else
{
	if(input[i]>(209+offset0))
{
	if(input[i]==(211+offset0))
{
	vR[i] = 211;
}
else
{
	vR[i] = 210;
}

}
else
{
	if(input[i]==(209+offset0))
{
	vR[i] = 209;
}
else
{
	vR[i] = 208;
}

}

}

}

}
else
{
	if(input[i]>(199+offset0))
{
	if(input[i]>(203+offset0))
{
	if(input[i]>(205+offset0))
{
	if(input[i]==(207+offset0))
{
	vR[i] = 207;
}
else
{
	vR[i] = 206;
}

}
else
{
	if(input[i]==(205+offset0))
{
	vR[i] = 205;
}
else
{
	vR[i] = 204;
}

}

}
else
{
	if(input[i]>(201+offset0))
{
	if(input[i]==(203+offset0))
{
	vR[i] = 203;
}
else
{
	vR[i] = 202;
}

}
else
{
	if(input[i]==(201+offset0))
{
	vR[i] = 201;
}
else
{
	vR[i] = 200;
}

}

}

}
else
{
	if(input[i]>(195+offset0))
{
	if(input[i]>(197+offset0))
{
	if(input[i]==(199+offset0))
{
	vR[i] = 199;
}
else
{
	vR[i] = 198;
}

}
else
{
	if(input[i]==(197+offset0))
{
	vR[i] = 197;
}
else
{
	vR[i] = 196;
}

}

}
else
{
	if(input[i]>(193+offset0))
{
	if(input[i]==(195+offset0))
{
	vR[i] = 195;
}
else
{
	vR[i] = 194;
}

}
else
{
	if(input[i]==(193+offset0))
{
	vR[i] = 193;
}
else
{
	vR[i] = 192;
}

}

}

}

}

}

}
else
{
	if(input[i]>(159+offset0))
{
	if(input[i]>(175+offset0))
{
	if(input[i]>(183+offset0))
{
	if(input[i]>(187+offset0))
{
	if(input[i]>(189+offset0))
{
	if(input[i]==(191+offset0))
{
	vR[i] = 191;
}
else
{
	vR[i] = 190;
}

}
else
{
	if(input[i]==(189+offset0))
{
	vR[i] = 189;
}
else
{
	vR[i] = 188;
}

}

}
else
{
	if(input[i]>(185+offset0))
{
	if(input[i]==(187+offset0))
{
	vR[i] = 187;
}
else
{
	vR[i] = 186;
}

}
else
{
	if(input[i]==(185+offset0))
{
	vR[i] = 185;
}
else
{
	vR[i] = 184;
}

}

}

}
else
{
	if(input[i]>(179+offset0))
{
	if(input[i]>(181+offset0))
{
	if(input[i]==(183+offset0))
{
	vR[i] = 183;
}
else
{
	vR[i] = 182;
}

}
else
{
	if(input[i]==(181+offset0))
{
	vR[i] = 181;
}
else
{
	vR[i] = 180;
}

}

}
else
{
	if(input[i]>(177+offset0))
{
	if(input[i]==(179+offset0))
{
	vR[i] = 179;
}
else
{
	vR[i] = 178;
}

}
else
{
	if(input[i]==(177+offset0))
{
	vR[i] = 177;
}
else
{
	vR[i] = 176;
}

}

}

}

}
else
{
	if(input[i]>(167+offset0))
{
	if(input[i]>(171+offset0))
{
	if(input[i]>(173+offset0))
{
	if(input[i]==(175+offset0))
{
	vR[i] = 175;
}
else
{
	vR[i] = 174;
}

}
else
{
	if(input[i]==(173+offset0))
{
	vR[i] = 173;
}
else
{
	vR[i] = 172;
}

}

}
else
{
	if(input[i]>(169+offset0))
{
	if(input[i]==(171+offset0))
{
	vR[i] = 171;
}
else
{
	vR[i] = 170;
}

}
else
{
	if(input[i]==(169+offset0))
{
	vR[i] = 169;
}
else
{
	vR[i] = 168;
}

}

}

}
else
{
	if(input[i]>(163+offset0))
{
	if(input[i]>(165+offset0))
{
	if(input[i]==(167+offset0))
{
	vR[i] = 167;
}
else
{
	vR[i] = 166;
}

}
else
{
	if(input[i]==(165+offset0))
{
	vR[i] = 165;
}
else
{
	vR[i] = 164;
}

}

}
else
{
	if(input[i]>(161+offset0))
{
	if(input[i]==(163+offset0))
{
	vR[i] = 163;
}
else
{
	vR[i] = 162;
}

}
else
{
	if(input[i]==(161+offset0))
{
	vR[i] = 161;
}
else
{
	vR[i] = 160;
}

}

}

}

}

}
else
{
	if(input[i]>(143+offset0))
{
	if(input[i]>(151+offset0))
{
	if(input[i]>(155+offset0))
{
	if(input[i]>(157+offset0))
{
	if(input[i]==(159+offset0))
{
	vR[i] = 159;
}
else
{
	vR[i] = 158;
}

}
else
{
	if(input[i]==(157+offset0))
{
	vR[i] = 157;
}
else
{
	vR[i] = 156;
}

}

}
else
{
	if(input[i]>(153+offset0))
{
	if(input[i]==(155+offset0))
{
	vR[i] = 155;
}
else
{
	vR[i] = 154;
}

}
else
{
	if(input[i]==(153+offset0))
{
	vR[i] = 153;
}
else
{
	vR[i] = 152;
}

}

}

}
else
{
	if(input[i]>(147+offset0))
{
	if(input[i]>(149+offset0))
{
	if(input[i]==(151+offset0))
{
	vR[i] = 151;
}
else
{
	vR[i] = 150;
}

}
else
{
	if(input[i]==(149+offset0))
{
	vR[i] = 149;
}
else
{
	vR[i] = 148;
}

}

}
else
{
	if(input[i]>(145+offset0))
{
	if(input[i]==(147+offset0))
{
	vR[i] = 147;
}
else
{
	vR[i] = 146;
}

}
else
{
	if(input[i]==(145+offset0))
{
	vR[i] = 145;
}
else
{
	vR[i] = 144;
}

}

}

}

}
else
{
	if(input[i]>(135+offset0))
{
	if(input[i]>(139+offset0))
{
	if(input[i]>(141+offset0))
{
	if(input[i]==(143+offset0))
{
	vR[i] = 143;
}
else
{
	vR[i] = 142;
}

}
else
{
	if(input[i]==(141+offset0))
{
	vR[i] = 141;
}
else
{
	vR[i] = 140;
}

}

}
else
{
	if(input[i]>(137+offset0))
{
	if(input[i]==(139+offset0))
{
	vR[i] = 139;
}
else
{
	vR[i] = 138;
}

}
else
{
	if(input[i]==(137+offset0))
{
	vR[i] = 137;
}
else
{
	vR[i] = 136;
}

}

}

}
else
{
	if(input[i]>(131+offset0))
{
	if(input[i]>(133+offset0))
{
	if(input[i]==(135+offset0))
{
	vR[i] = 135;
}
else
{
	vR[i] = 134;
}

}
else
{
	if(input[i]==(133+offset0))
{
	vR[i] = 133;
}
else
{
	vR[i] = 132;
}

}

}
else
{
	if(input[i]>(129+offset0))
{
	if(input[i]==(131+offset0))
{
	vR[i] = 131;
}
else
{
	vR[i] = 130;
}

}
else
{
	if(input[i]==(129+offset0))
{
	vR[i] = 129;
}
else
{
	vR[i] = 128;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(63+offset0))
{
	if(input[i]>(95+offset0))
{
	if(input[i]>(111+offset0))
{
	if(input[i]>(119+offset0))
{
	if(input[i]>(123+offset0))
{
	if(input[i]>(125+offset0))
{
	if(input[i]==(127+offset0))
{
	vR[i] = 127;
}
else
{
	vR[i] = 126;
}

}
else
{
	if(input[i]==(125+offset0))
{
	vR[i] = 125;
}
else
{
	vR[i] = 124;
}

}

}
else
{
	if(input[i]>(121+offset0))
{
	if(input[i]==(123+offset0))
{
	vR[i] = 123;
}
else
{
	vR[i] = 122;
}

}
else
{
	if(input[i]==(121+offset0))
{
	vR[i] = 121;
}
else
{
	vR[i] = 120;
}

}

}

}
else
{
	if(input[i]>(115+offset0))
{
	if(input[i]>(117+offset0))
{
	if(input[i]==(119+offset0))
{
	vR[i] = 119;
}
else
{
	vR[i] = 118;
}

}
else
{
	if(input[i]==(117+offset0))
{
	vR[i] = 117;
}
else
{
	vR[i] = 116;
}

}

}
else
{
	if(input[i]>(113+offset0))
{
	if(input[i]==(115+offset0))
{
	vR[i] = 115;
}
else
{
	vR[i] = 114;
}

}
else
{
	if(input[i]==(113+offset0))
{
	vR[i] = 113;
}
else
{
	vR[i] = 112;
}

}

}

}

}
else
{
	if(input[i]>(103+offset0))
{
	if(input[i]>(107+offset0))
{
	if(input[i]>(109+offset0))
{
	if(input[i]==(111+offset0))
{
	vR[i] = 111;
}
else
{
	vR[i] = 110;
}

}
else
{
	if(input[i]==(109+offset0))
{
	vR[i] = 109;
}
else
{
	vR[i] = 108;
}

}

}
else
{
	if(input[i]>(105+offset0))
{
	if(input[i]==(107+offset0))
{
	vR[i] = 107;
}
else
{
	vR[i] = 106;
}

}
else
{
	if(input[i]==(105+offset0))
{
	vR[i] = 105;
}
else
{
	vR[i] = 104;
}

}

}

}
else
{
	if(input[i]>(99+offset0))
{
	if(input[i]>(101+offset0))
{
	if(input[i]==(103+offset0))
{
	vR[i] = 103;
}
else
{
	vR[i] = 102;
}

}
else
{
	if(input[i]==(101+offset0))
{
	vR[i] = 101;
}
else
{
	vR[i] = 100;
}

}

}
else
{
	if(input[i]>(97+offset0))
{
	if(input[i]==(99+offset0))
{
	vR[i] = 99;
}
else
{
	vR[i] = 98;
}

}
else
{
	if(input[i]==(97+offset0))
{
	vR[i] = 97;
}
else
{
	vR[i] = 96;
}

}

}

}

}

}
else
{
	if(input[i]>(79+offset0))
{
	if(input[i]>(87+offset0))
{
	if(input[i]>(91+offset0))
{
	if(input[i]>(93+offset0))
{
	if(input[i]==(95+offset0))
{
	vR[i] = 95;
}
else
{
	vR[i] = 94;
}

}
else
{
	if(input[i]==(93+offset0))
{
	vR[i] = 93;
}
else
{
	vR[i] = 92;
}

}

}
else
{
	if(input[i]>(89+offset0))
{
	if(input[i]==(91+offset0))
{
	vR[i] = 91;
}
else
{
	vR[i] = 90;
}

}
else
{
	if(input[i]==(89+offset0))
{
	vR[i] = 89;
}
else
{
	vR[i] = 88;
}

}

}

}
else
{
	if(input[i]>(83+offset0))
{
	if(input[i]>(85+offset0))
{
	if(input[i]==(87+offset0))
{
	vR[i] = 87;
}
else
{
	vR[i] = 86;
}

}
else
{
	if(input[i]==(85+offset0))
{
	vR[i] = 85;
}
else
{
	vR[i] = 84;
}

}

}
else
{
	if(input[i]>(81+offset0))
{
	if(input[i]==(83+offset0))
{
	vR[i] = 83;
}
else
{
	vR[i] = 82;
}

}
else
{
	if(input[i]==(81+offset0))
{
	vR[i] = 81;
}
else
{
	vR[i] = 80;
}

}

}

}

}
else
{
	if(input[i]>(71+offset0))
{
	if(input[i]>(75+offset0))
{
	if(input[i]>(77+offset0))
{
	if(input[i]==(79+offset0))
{
	vR[i] = 79;
}
else
{
	vR[i] = 78;
}

}
else
{
	if(input[i]==(77+offset0))
{
	vR[i] = 77;
}
else
{
	vR[i] = 76;
}

}

}
else
{
	if(input[i]>(73+offset0))
{
	if(input[i]==(75+offset0))
{
	vR[i] = 75;
}
else
{
	vR[i] = 74;
}

}
else
{
	if(input[i]==(73+offset0))
{
	vR[i] = 73;
}
else
{
	vR[i] = 72;
}

}

}

}
else
{
	if(input[i]>(67+offset0))
{
	if(input[i]>(69+offset0))
{
	if(input[i]==(71+offset0))
{
	vR[i] = 71;
}
else
{
	vR[i] = 70;
}

}
else
{
	if(input[i]==(69+offset0))
{
	vR[i] = 69;
}
else
{
	vR[i] = 68;
}

}

}
else
{
	if(input[i]>(65+offset0))
{
	if(input[i]==(67+offset0))
{
	vR[i] = 67;
}
else
{
	vR[i] = 66;
}

}
else
{
	if(input[i]==(65+offset0))
{
	vR[i] = 65;
}
else
{
	vR[i] = 64;
}

}

}

}

}

}

}
else
{
	if(input[i]>(31+offset0))
{
	if(input[i]>(47+offset0))
{
	if(input[i]>(55+offset0))
{
	if(input[i]>(59+offset0))
{
	if(input[i]>(61+offset0))
{
	if(input[i]==(63+offset0))
{
	vR[i] = 63;
}
else
{
	vR[i] = 62;
}

}
else
{
	if(input[i]==(61+offset0))
{
	vR[i] = 61;
}
else
{
	vR[i] = 60;
}

}

}
else
{
	if(input[i]>(57+offset0))
{
	if(input[i]==(59+offset0))
{
	vR[i] = 59;
}
else
{
	vR[i] = 58;
}

}
else
{
	if(input[i]==(57+offset0))
{
	vR[i] = 57;
}
else
{
	vR[i] = 56;
}

}

}

}
else
{
	if(input[i]>(51+offset0))
{
	if(input[i]>(53+offset0))
{
	if(input[i]==(55+offset0))
{
	vR[i] = 55;
}
else
{
	vR[i] = 54;
}

}
else
{
	if(input[i]==(53+offset0))
{
	vR[i] = 53;
}
else
{
	vR[i] = 52;
}

}

}
else
{
	if(input[i]>(49+offset0))
{
	if(input[i]==(51+offset0))
{
	vR[i] = 51;
}
else
{
	vR[i] = 50;
}

}
else
{
	if(input[i]==(49+offset0))
{
	vR[i] = 49;
}
else
{
	vR[i] = 48;
}

}

}

}

}
else
{
	if(input[i]>(39+offset0))
{
	if(input[i]>(43+offset0))
{
	if(input[i]>(45+offset0))
{
	if(input[i]==(47+offset0))
{
	vR[i] = 47;
}
else
{
	vR[i] = 46;
}

}
else
{
	if(input[i]==(45+offset0))
{
	vR[i] = 45;
}
else
{
	vR[i] = 44;
}

}

}
else
{
	if(input[i]>(41+offset0))
{
	if(input[i]==(43+offset0))
{
	vR[i] = 43;
}
else
{
	vR[i] = 42;
}

}
else
{
	if(input[i]==(41+offset0))
{
	vR[i] = 41;
}
else
{
	vR[i] = 40;
}

}

}

}
else
{
	if(input[i]>(35+offset0))
{
	if(input[i]>(37+offset0))
{
	if(input[i]==(39+offset0))
{
	vR[i] = 39;
}
else
{
	vR[i] = 38;
}

}
else
{
	if(input[i]==(37+offset0))
{
	vR[i] = 37;
}
else
{
	vR[i] = 36;
}

}

}
else
{
	if(input[i]>(33+offset0))
{
	if(input[i]==(35+offset0))
{
	vR[i] = 35;
}
else
{
	vR[i] = 34;
}

}
else
{
	if(input[i]==(33+offset0))
{
	vR[i] = 33;
}
else
{
	vR[i] = 32;
}

}

}

}

}

}
else
{
	if(input[i]>(15+offset0))
{
	if(input[i]>(23+offset0))
{
	if(input[i]>(27+offset0))
{
	if(input[i]>(29+offset0))
{
	if(input[i]==(31+offset0))
{
	vR[i] = 31;
}
else
{
	vR[i] = 30;
}

}
else
{
	if(input[i]==(29+offset0))
{
	vR[i] = 29;
}
else
{
	vR[i] = 28;
}

}

}
else
{
	if(input[i]>(25+offset0))
{
	if(input[i]==(27+offset0))
{
	vR[i] = 27;
}
else
{
	vR[i] = 26;
}

}
else
{
	if(input[i]==(25+offset0))
{
	vR[i] = 25;
}
else
{
	vR[i] = 24;
}

}

}

}
else
{
	if(input[i]>(19+offset0))
{
	if(input[i]>(21+offset0))
{
	if(input[i]==(23+offset0))
{
	vR[i] = 23;
}
else
{
	vR[i] = 22;
}

}
else
{
	if(input[i]==(21+offset0))
{
	vR[i] = 21;
}
else
{
	vR[i] = 20;
}

}

}
else
{
	if(input[i]>(17+offset0))
{
	if(input[i]==(19+offset0))
{
	vR[i] = 19;
}
else
{
	vR[i] = 18;
}

}
else
{
	if(input[i]==(17+offset0))
{
	vR[i] = 17;
}
else
{
	vR[i] = 16;
}

}

}

}

}
else
{
	if(input[i]>(7+offset0))
{
	if(input[i]>(11+offset0))
{
	if(input[i]>(13+offset0))
{
	if(input[i]==(15+offset0))
{
	vR[i] = 15;
}
else
{
	vR[i] = 14;
}

}
else
{
	if(input[i]==(13+offset0))
{
	vR[i] = 13;
}
else
{
	vR[i] = 12;
}

}

}
else
{
	if(input[i]>(9+offset0))
{
	if(input[i]==(11+offset0))
{
	vR[i] = 11;
}
else
{
	vR[i] = 10;
}

}
else
{
	if(input[i]==(9+offset0))
{
	vR[i] = 9;
}
else
{
	vR[i] = 8;
}

}

}

}
else
{
	if(input[i]>(3+offset0))
{
	if(input[i]>(5+offset0))
{
	if(input[i]==(7+offset0))
{
	vR[i] = 7;
}
else
{
	vR[i] = 6;
}

}
else
{
	if(input[i]==(5+offset0))
{
	vR[i] = 5;
}
else
{
	vR[i] = 4;
}

}

}
else
{
	if(input[i]>(1+offset0))
{
	if(input[i]==(3+offset0))
{
	vR[i] = 3;
}
else
{
	vR[i] = 2;
}

}
else
{
	if(input[i]==(1+offset0))
{
	vR[i] = 1;
}
else
{
	vR[i] = 0;
}

}

}

}

}

}

}

}

}

}

}


if(input[i]>(1023+offset1))
{
	if(input[i]>(1535+offset1))
{
	if(input[i]>(1791+offset1))
{
	if(input[i]>(1919+offset1))
{
	if(input[i]>(1983+offset1))
{
	if(input[i]>(2015+offset1))
{
	if(input[i]>(2031+offset1))
{
	if(input[i]>(2039+offset1))
{
	if(input[i]>(2043+offset1))
{
	if(input[i]>(2045+offset1))
{
	if(input[i]==(2047+offset1))
{
	vR[i] = 2047;
}
else
{
	vR[i] = 2046;
}

}
else
{
	if(input[i]==(2045+offset1))
{
	vR[i] = 2045;
}
else
{
	vR[i] = 2044;
}

}

}
else
{
	if(input[i]>(2041+offset1))
{
	if(input[i]==(2043+offset1))
{
	vR[i] = 2043;
}
else
{
	vR[i] = 2042;
}

}
else
{
	if(input[i]==(2041+offset1))
{
	vR[i] = 2041;
}
else
{
	vR[i] = 2040;
}

}

}

}
else
{
	if(input[i]>(2035+offset1))
{
	if(input[i]>(2037+offset1))
{
	if(input[i]==(2039+offset1))
{
	vR[i] = 2039;
}
else
{
	vR[i] = 2038;
}

}
else
{
	if(input[i]==(2037+offset1))
{
	vR[i] = 2037;
}
else
{
	vR[i] = 2036;
}

}

}
else
{
	if(input[i]>(2033+offset1))
{
	if(input[i]==(2035+offset1))
{
	vR[i] = 2035;
}
else
{
	vR[i] = 2034;
}

}
else
{
	if(input[i]==(2033+offset1))
{
	vR[i] = 2033;
}
else
{
	vR[i] = 2032;
}

}

}

}

}
else
{
	if(input[i]>(2023+offset1))
{
	if(input[i]>(2027+offset1))
{
	if(input[i]>(2029+offset1))
{
	if(input[i]==(2031+offset1))
{
	vR[i] = 2031;
}
else
{
	vR[i] = 2030;
}

}
else
{
	if(input[i]==(2029+offset1))
{
	vR[i] = 2029;
}
else
{
	vR[i] = 2028;
}

}

}
else
{
	if(input[i]>(2025+offset1))
{
	if(input[i]==(2027+offset1))
{
	vR[i] = 2027;
}
else
{
	vR[i] = 2026;
}

}
else
{
	if(input[i]==(2025+offset1))
{
	vR[i] = 2025;
}
else
{
	vR[i] = 2024;
}

}

}

}
else
{
	if(input[i]>(2019+offset1))
{
	if(input[i]>(2021+offset1))
{
	if(input[i]==(2023+offset1))
{
	vR[i] = 2023;
}
else
{
	vR[i] = 2022;
}

}
else
{
	if(input[i]==(2021+offset1))
{
	vR[i] = 2021;
}
else
{
	vR[i] = 2020;
}

}

}
else
{
	if(input[i]>(2017+offset1))
{
	if(input[i]==(2019+offset1))
{
	vR[i] = 2019;
}
else
{
	vR[i] = 2018;
}

}
else
{
	if(input[i]==(2017+offset1))
{
	vR[i] = 2017;
}
else
{
	vR[i] = 2016;
}

}

}

}

}

}
else
{
	if(input[i]>(1999+offset1))
{
	if(input[i]>(2007+offset1))
{
	if(input[i]>(2011+offset1))
{
	if(input[i]>(2013+offset1))
{
	if(input[i]==(2015+offset1))
{
	vR[i] = 2015;
}
else
{
	vR[i] = 2014;
}

}
else
{
	if(input[i]==(2013+offset1))
{
	vR[i] = 2013;
}
else
{
	vR[i] = 2012;
}

}

}
else
{
	if(input[i]>(2009+offset1))
{
	if(input[i]==(2011+offset1))
{
	vR[i] = 2011;
}
else
{
	vR[i] = 2010;
}

}
else
{
	if(input[i]==(2009+offset1))
{
	vR[i] = 2009;
}
else
{
	vR[i] = 2008;
}

}

}

}
else
{
	if(input[i]>(2003+offset1))
{
	if(input[i]>(2005+offset1))
{
	if(input[i]==(2007+offset1))
{
	vR[i] = 2007;
}
else
{
	vR[i] = 2006;
}

}
else
{
	if(input[i]==(2005+offset1))
{
	vR[i] = 2005;
}
else
{
	vR[i] = 2004;
}

}

}
else
{
	if(input[i]>(2001+offset1))
{
	if(input[i]==(2003+offset1))
{
	vR[i] = 2003;
}
else
{
	vR[i] = 2002;
}

}
else
{
	if(input[i]==(2001+offset1))
{
	vR[i] = 2001;
}
else
{
	vR[i] = 2000;
}

}

}

}

}
else
{
	if(input[i]>(1991+offset1))
{
	if(input[i]>(1995+offset1))
{
	if(input[i]>(1997+offset1))
{
	if(input[i]==(1999+offset1))
{
	vR[i] = 1999;
}
else
{
	vR[i] = 1998;
}

}
else
{
	if(input[i]==(1997+offset1))
{
	vR[i] = 1997;
}
else
{
	vR[i] = 1996;
}

}

}
else
{
	if(input[i]>(1993+offset1))
{
	if(input[i]==(1995+offset1))
{
	vR[i] = 1995;
}
else
{
	vR[i] = 1994;
}

}
else
{
	if(input[i]==(1993+offset1))
{
	vR[i] = 1993;
}
else
{
	vR[i] = 1992;
}

}

}

}
else
{
	if(input[i]>(1987+offset1))
{
	if(input[i]>(1989+offset1))
{
	if(input[i]==(1991+offset1))
{
	vR[i] = 1991;
}
else
{
	vR[i] = 1990;
}

}
else
{
	if(input[i]==(1989+offset1))
{
	vR[i] = 1989;
}
else
{
	vR[i] = 1988;
}

}

}
else
{
	if(input[i]>(1985+offset1))
{
	if(input[i]==(1987+offset1))
{
	vR[i] = 1987;
}
else
{
	vR[i] = 1986;
}

}
else
{
	if(input[i]==(1985+offset1))
{
	vR[i] = 1985;
}
else
{
	vR[i] = 1984;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1951+offset1))
{
	if(input[i]>(1967+offset1))
{
	if(input[i]>(1975+offset1))
{
	if(input[i]>(1979+offset1))
{
	if(input[i]>(1981+offset1))
{
	if(input[i]==(1983+offset1))
{
	vR[i] = 1983;
}
else
{
	vR[i] = 1982;
}

}
else
{
	if(input[i]==(1981+offset1))
{
	vR[i] = 1981;
}
else
{
	vR[i] = 1980;
}

}

}
else
{
	if(input[i]>(1977+offset1))
{
	if(input[i]==(1979+offset1))
{
	vR[i] = 1979;
}
else
{
	vR[i] = 1978;
}

}
else
{
	if(input[i]==(1977+offset1))
{
	vR[i] = 1977;
}
else
{
	vR[i] = 1976;
}

}

}

}
else
{
	if(input[i]>(1971+offset1))
{
	if(input[i]>(1973+offset1))
{
	if(input[i]==(1975+offset1))
{
	vR[i] = 1975;
}
else
{
	vR[i] = 1974;
}

}
else
{
	if(input[i]==(1973+offset1))
{
	vR[i] = 1973;
}
else
{
	vR[i] = 1972;
}

}

}
else
{
	if(input[i]>(1969+offset1))
{
	if(input[i]==(1971+offset1))
{
	vR[i] = 1971;
}
else
{
	vR[i] = 1970;
}

}
else
{
	if(input[i]==(1969+offset1))
{
	vR[i] = 1969;
}
else
{
	vR[i] = 1968;
}

}

}

}

}
else
{
	if(input[i]>(1959+offset1))
{
	if(input[i]>(1963+offset1))
{
	if(input[i]>(1965+offset1))
{
	if(input[i]==(1967+offset1))
{
	vR[i] = 1967;
}
else
{
	vR[i] = 1966;
}

}
else
{
	if(input[i]==(1965+offset1))
{
	vR[i] = 1965;
}
else
{
	vR[i] = 1964;
}

}

}
else
{
	if(input[i]>(1961+offset1))
{
	if(input[i]==(1963+offset1))
{
	vR[i] = 1963;
}
else
{
	vR[i] = 1962;
}

}
else
{
	if(input[i]==(1961+offset1))
{
	vR[i] = 1961;
}
else
{
	vR[i] = 1960;
}

}

}

}
else
{
	if(input[i]>(1955+offset1))
{
	if(input[i]>(1957+offset1))
{
	if(input[i]==(1959+offset1))
{
	vR[i] = 1959;
}
else
{
	vR[i] = 1958;
}

}
else
{
	if(input[i]==(1957+offset1))
{
	vR[i] = 1957;
}
else
{
	vR[i] = 1956;
}

}

}
else
{
	if(input[i]>(1953+offset1))
{
	if(input[i]==(1955+offset1))
{
	vR[i] = 1955;
}
else
{
	vR[i] = 1954;
}

}
else
{
	if(input[i]==(1953+offset1))
{
	vR[i] = 1953;
}
else
{
	vR[i] = 1952;
}

}

}

}

}

}
else
{
	if(input[i]>(1935+offset1))
{
	if(input[i]>(1943+offset1))
{
	if(input[i]>(1947+offset1))
{
	if(input[i]>(1949+offset1))
{
	if(input[i]==(1951+offset1))
{
	vR[i] = 1951;
}
else
{
	vR[i] = 1950;
}

}
else
{
	if(input[i]==(1949+offset1))
{
	vR[i] = 1949;
}
else
{
	vR[i] = 1948;
}

}

}
else
{
	if(input[i]>(1945+offset1))
{
	if(input[i]==(1947+offset1))
{
	vR[i] = 1947;
}
else
{
	vR[i] = 1946;
}

}
else
{
	if(input[i]==(1945+offset1))
{
	vR[i] = 1945;
}
else
{
	vR[i] = 1944;
}

}

}

}
else
{
	if(input[i]>(1939+offset1))
{
	if(input[i]>(1941+offset1))
{
	if(input[i]==(1943+offset1))
{
	vR[i] = 1943;
}
else
{
	vR[i] = 1942;
}

}
else
{
	if(input[i]==(1941+offset1))
{
	vR[i] = 1941;
}
else
{
	vR[i] = 1940;
}

}

}
else
{
	if(input[i]>(1937+offset1))
{
	if(input[i]==(1939+offset1))
{
	vR[i] = 1939;
}
else
{
	vR[i] = 1938;
}

}
else
{
	if(input[i]==(1937+offset1))
{
	vR[i] = 1937;
}
else
{
	vR[i] = 1936;
}

}

}

}

}
else
{
	if(input[i]>(1927+offset1))
{
	if(input[i]>(1931+offset1))
{
	if(input[i]>(1933+offset1))
{
	if(input[i]==(1935+offset1))
{
	vR[i] = 1935;
}
else
{
	vR[i] = 1934;
}

}
else
{
	if(input[i]==(1933+offset1))
{
	vR[i] = 1933;
}
else
{
	vR[i] = 1932;
}

}

}
else
{
	if(input[i]>(1929+offset1))
{
	if(input[i]==(1931+offset1))
{
	vR[i] = 1931;
}
else
{
	vR[i] = 1930;
}

}
else
{
	if(input[i]==(1929+offset1))
{
	vR[i] = 1929;
}
else
{
	vR[i] = 1928;
}

}

}

}
else
{
	if(input[i]>(1923+offset1))
{
	if(input[i]>(1925+offset1))
{
	if(input[i]==(1927+offset1))
{
	vR[i] = 1927;
}
else
{
	vR[i] = 1926;
}

}
else
{
	if(input[i]==(1925+offset1))
{
	vR[i] = 1925;
}
else
{
	vR[i] = 1924;
}

}

}
else
{
	if(input[i]>(1921+offset1))
{
	if(input[i]==(1923+offset1))
{
	vR[i] = 1923;
}
else
{
	vR[i] = 1922;
}

}
else
{
	if(input[i]==(1921+offset1))
{
	vR[i] = 1921;
}
else
{
	vR[i] = 1920;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(1855+offset1))
{
	if(input[i]>(1887+offset1))
{
	if(input[i]>(1903+offset1))
{
	if(input[i]>(1911+offset1))
{
	if(input[i]>(1915+offset1))
{
	if(input[i]>(1917+offset1))
{
	if(input[i]==(1919+offset1))
{
	vR[i] = 1919;
}
else
{
	vR[i] = 1918;
}

}
else
{
	if(input[i]==(1917+offset1))
{
	vR[i] = 1917;
}
else
{
	vR[i] = 1916;
}

}

}
else
{
	if(input[i]>(1913+offset1))
{
	if(input[i]==(1915+offset1))
{
	vR[i] = 1915;
}
else
{
	vR[i] = 1914;
}

}
else
{
	if(input[i]==(1913+offset1))
{
	vR[i] = 1913;
}
else
{
	vR[i] = 1912;
}

}

}

}
else
{
	if(input[i]>(1907+offset1))
{
	if(input[i]>(1909+offset1))
{
	if(input[i]==(1911+offset1))
{
	vR[i] = 1911;
}
else
{
	vR[i] = 1910;
}

}
else
{
	if(input[i]==(1909+offset1))
{
	vR[i] = 1909;
}
else
{
	vR[i] = 1908;
}

}

}
else
{
	if(input[i]>(1905+offset1))
{
	if(input[i]==(1907+offset1))
{
	vR[i] = 1907;
}
else
{
	vR[i] = 1906;
}

}
else
{
	if(input[i]==(1905+offset1))
{
	vR[i] = 1905;
}
else
{
	vR[i] = 1904;
}

}

}

}

}
else
{
	if(input[i]>(1895+offset1))
{
	if(input[i]>(1899+offset1))
{
	if(input[i]>(1901+offset1))
{
	if(input[i]==(1903+offset1))
{
	vR[i] = 1903;
}
else
{
	vR[i] = 1902;
}

}
else
{
	if(input[i]==(1901+offset1))
{
	vR[i] = 1901;
}
else
{
	vR[i] = 1900;
}

}

}
else
{
	if(input[i]>(1897+offset1))
{
	if(input[i]==(1899+offset1))
{
	vR[i] = 1899;
}
else
{
	vR[i] = 1898;
}

}
else
{
	if(input[i]==(1897+offset1))
{
	vR[i] = 1897;
}
else
{
	vR[i] = 1896;
}

}

}

}
else
{
	if(input[i]>(1891+offset1))
{
	if(input[i]>(1893+offset1))
{
	if(input[i]==(1895+offset1))
{
	vR[i] = 1895;
}
else
{
	vR[i] = 1894;
}

}
else
{
	if(input[i]==(1893+offset1))
{
	vR[i] = 1893;
}
else
{
	vR[i] = 1892;
}

}

}
else
{
	if(input[i]>(1889+offset1))
{
	if(input[i]==(1891+offset1))
{
	vR[i] = 1891;
}
else
{
	vR[i] = 1890;
}

}
else
{
	if(input[i]==(1889+offset1))
{
	vR[i] = 1889;
}
else
{
	vR[i] = 1888;
}

}

}

}

}

}
else
{
	if(input[i]>(1871+offset1))
{
	if(input[i]>(1879+offset1))
{
	if(input[i]>(1883+offset1))
{
	if(input[i]>(1885+offset1))
{
	if(input[i]==(1887+offset1))
{
	vR[i] = 1887;
}
else
{
	vR[i] = 1886;
}

}
else
{
	if(input[i]==(1885+offset1))
{
	vR[i] = 1885;
}
else
{
	vR[i] = 1884;
}

}

}
else
{
	if(input[i]>(1881+offset1))
{
	if(input[i]==(1883+offset1))
{
	vR[i] = 1883;
}
else
{
	vR[i] = 1882;
}

}
else
{
	if(input[i]==(1881+offset1))
{
	vR[i] = 1881;
}
else
{
	vR[i] = 1880;
}

}

}

}
else
{
	if(input[i]>(1875+offset1))
{
	if(input[i]>(1877+offset1))
{
	if(input[i]==(1879+offset1))
{
	vR[i] = 1879;
}
else
{
	vR[i] = 1878;
}

}
else
{
	if(input[i]==(1877+offset1))
{
	vR[i] = 1877;
}
else
{
	vR[i] = 1876;
}

}

}
else
{
	if(input[i]>(1873+offset1))
{
	if(input[i]==(1875+offset1))
{
	vR[i] = 1875;
}
else
{
	vR[i] = 1874;
}

}
else
{
	if(input[i]==(1873+offset1))
{
	vR[i] = 1873;
}
else
{
	vR[i] = 1872;
}

}

}

}

}
else
{
	if(input[i]>(1863+offset1))
{
	if(input[i]>(1867+offset1))
{
	if(input[i]>(1869+offset1))
{
	if(input[i]==(1871+offset1))
{
	vR[i] = 1871;
}
else
{
	vR[i] = 1870;
}

}
else
{
	if(input[i]==(1869+offset1))
{
	vR[i] = 1869;
}
else
{
	vR[i] = 1868;
}

}

}
else
{
	if(input[i]>(1865+offset1))
{
	if(input[i]==(1867+offset1))
{
	vR[i] = 1867;
}
else
{
	vR[i] = 1866;
}

}
else
{
	if(input[i]==(1865+offset1))
{
	vR[i] = 1865;
}
else
{
	vR[i] = 1864;
}

}

}

}
else
{
	if(input[i]>(1859+offset1))
{
	if(input[i]>(1861+offset1))
{
	if(input[i]==(1863+offset1))
{
	vR[i] = 1863;
}
else
{
	vR[i] = 1862;
}

}
else
{
	if(input[i]==(1861+offset1))
{
	vR[i] = 1861;
}
else
{
	vR[i] = 1860;
}

}

}
else
{
	if(input[i]>(1857+offset1))
{
	if(input[i]==(1859+offset1))
{
	vR[i] = 1859;
}
else
{
	vR[i] = 1858;
}

}
else
{
	if(input[i]==(1857+offset1))
{
	vR[i] = 1857;
}
else
{
	vR[i] = 1856;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1823+offset1))
{
	if(input[i]>(1839+offset1))
{
	if(input[i]>(1847+offset1))
{
	if(input[i]>(1851+offset1))
{
	if(input[i]>(1853+offset1))
{
	if(input[i]==(1855+offset1))
{
	vR[i] = 1855;
}
else
{
	vR[i] = 1854;
}

}
else
{
	if(input[i]==(1853+offset1))
{
	vR[i] = 1853;
}
else
{
	vR[i] = 1852;
}

}

}
else
{
	if(input[i]>(1849+offset1))
{
	if(input[i]==(1851+offset1))
{
	vR[i] = 1851;
}
else
{
	vR[i] = 1850;
}

}
else
{
	if(input[i]==(1849+offset1))
{
	vR[i] = 1849;
}
else
{
	vR[i] = 1848;
}

}

}

}
else
{
	if(input[i]>(1843+offset1))
{
	if(input[i]>(1845+offset1))
{
	if(input[i]==(1847+offset1))
{
	vR[i] = 1847;
}
else
{
	vR[i] = 1846;
}

}
else
{
	if(input[i]==(1845+offset1))
{
	vR[i] = 1845;
}
else
{
	vR[i] = 1844;
}

}

}
else
{
	if(input[i]>(1841+offset1))
{
	if(input[i]==(1843+offset1))
{
	vR[i] = 1843;
}
else
{
	vR[i] = 1842;
}

}
else
{
	if(input[i]==(1841+offset1))
{
	vR[i] = 1841;
}
else
{
	vR[i] = 1840;
}

}

}

}

}
else
{
	if(input[i]>(1831+offset1))
{
	if(input[i]>(1835+offset1))
{
	if(input[i]>(1837+offset1))
{
	if(input[i]==(1839+offset1))
{
	vR[i] = 1839;
}
else
{
	vR[i] = 1838;
}

}
else
{
	if(input[i]==(1837+offset1))
{
	vR[i] = 1837;
}
else
{
	vR[i] = 1836;
}

}

}
else
{
	if(input[i]>(1833+offset1))
{
	if(input[i]==(1835+offset1))
{
	vR[i] = 1835;
}
else
{
	vR[i] = 1834;
}

}
else
{
	if(input[i]==(1833+offset1))
{
	vR[i] = 1833;
}
else
{
	vR[i] = 1832;
}

}

}

}
else
{
	if(input[i]>(1827+offset1))
{
	if(input[i]>(1829+offset1))
{
	if(input[i]==(1831+offset1))
{
	vR[i] = 1831;
}
else
{
	vR[i] = 1830;
}

}
else
{
	if(input[i]==(1829+offset1))
{
	vR[i] = 1829;
}
else
{
	vR[i] = 1828;
}

}

}
else
{
	if(input[i]>(1825+offset1))
{
	if(input[i]==(1827+offset1))
{
	vR[i] = 1827;
}
else
{
	vR[i] = 1826;
}

}
else
{
	if(input[i]==(1825+offset1))
{
	vR[i] = 1825;
}
else
{
	vR[i] = 1824;
}

}

}

}

}

}
else
{
	if(input[i]>(1807+offset1))
{
	if(input[i]>(1815+offset1))
{
	if(input[i]>(1819+offset1))
{
	if(input[i]>(1821+offset1))
{
	if(input[i]==(1823+offset1))
{
	vR[i] = 1823;
}
else
{
	vR[i] = 1822;
}

}
else
{
	if(input[i]==(1821+offset1))
{
	vR[i] = 1821;
}
else
{
	vR[i] = 1820;
}

}

}
else
{
	if(input[i]>(1817+offset1))
{
	if(input[i]==(1819+offset1))
{
	vR[i] = 1819;
}
else
{
	vR[i] = 1818;
}

}
else
{
	if(input[i]==(1817+offset1))
{
	vR[i] = 1817;
}
else
{
	vR[i] = 1816;
}

}

}

}
else
{
	if(input[i]>(1811+offset1))
{
	if(input[i]>(1813+offset1))
{
	if(input[i]==(1815+offset1))
{
	vR[i] = 1815;
}
else
{
	vR[i] = 1814;
}

}
else
{
	if(input[i]==(1813+offset1))
{
	vR[i] = 1813;
}
else
{
	vR[i] = 1812;
}

}

}
else
{
	if(input[i]>(1809+offset1))
{
	if(input[i]==(1811+offset1))
{
	vR[i] = 1811;
}
else
{
	vR[i] = 1810;
}

}
else
{
	if(input[i]==(1809+offset1))
{
	vR[i] = 1809;
}
else
{
	vR[i] = 1808;
}

}

}

}

}
else
{
	if(input[i]>(1799+offset1))
{
	if(input[i]>(1803+offset1))
{
	if(input[i]>(1805+offset1))
{
	if(input[i]==(1807+offset1))
{
	vR[i] = 1807;
}
else
{
	vR[i] = 1806;
}

}
else
{
	if(input[i]==(1805+offset1))
{
	vR[i] = 1805;
}
else
{
	vR[i] = 1804;
}

}

}
else
{
	if(input[i]>(1801+offset1))
{
	if(input[i]==(1803+offset1))
{
	vR[i] = 1803;
}
else
{
	vR[i] = 1802;
}

}
else
{
	if(input[i]==(1801+offset1))
{
	vR[i] = 1801;
}
else
{
	vR[i] = 1800;
}

}

}

}
else
{
	if(input[i]>(1795+offset1))
{
	if(input[i]>(1797+offset1))
{
	if(input[i]==(1799+offset1))
{
	vR[i] = 1799;
}
else
{
	vR[i] = 1798;
}

}
else
{
	if(input[i]==(1797+offset1))
{
	vR[i] = 1797;
}
else
{
	vR[i] = 1796;
}

}

}
else
{
	if(input[i]>(1793+offset1))
{
	if(input[i]==(1795+offset1))
{
	vR[i] = 1795;
}
else
{
	vR[i] = 1794;
}

}
else
{
	if(input[i]==(1793+offset1))
{
	vR[i] = 1793;
}
else
{
	vR[i] = 1792;
}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(1663+offset1))
{
	if(input[i]>(1727+offset1))
{
	if(input[i]>(1759+offset1))
{
	if(input[i]>(1775+offset1))
{
	if(input[i]>(1783+offset1))
{
	if(input[i]>(1787+offset1))
{
	if(input[i]>(1789+offset1))
{
	if(input[i]==(1791+offset1))
{
	vR[i] = 1791;
}
else
{
	vR[i] = 1790;
}

}
else
{
	if(input[i]==(1789+offset1))
{
	vR[i] = 1789;
}
else
{
	vR[i] = 1788;
}

}

}
else
{
	if(input[i]>(1785+offset1))
{
	if(input[i]==(1787+offset1))
{
	vR[i] = 1787;
}
else
{
	vR[i] = 1786;
}

}
else
{
	if(input[i]==(1785+offset1))
{
	vR[i] = 1785;
}
else
{
	vR[i] = 1784;
}

}

}

}
else
{
	if(input[i]>(1779+offset1))
{
	if(input[i]>(1781+offset1))
{
	if(input[i]==(1783+offset1))
{
	vR[i] = 1783;
}
else
{
	vR[i] = 1782;
}

}
else
{
	if(input[i]==(1781+offset1))
{
	vR[i] = 1781;
}
else
{
	vR[i] = 1780;
}

}

}
else
{
	if(input[i]>(1777+offset1))
{
	if(input[i]==(1779+offset1))
{
	vR[i] = 1779;
}
else
{
	vR[i] = 1778;
}

}
else
{
	if(input[i]==(1777+offset1))
{
	vR[i] = 1777;
}
else
{
	vR[i] = 1776;
}

}

}

}

}
else
{
	if(input[i]>(1767+offset1))
{
	if(input[i]>(1771+offset1))
{
	if(input[i]>(1773+offset1))
{
	if(input[i]==(1775+offset1))
{
	vR[i] = 1775;
}
else
{
	vR[i] = 1774;
}

}
else
{
	if(input[i]==(1773+offset1))
{
	vR[i] = 1773;
}
else
{
	vR[i] = 1772;
}

}

}
else
{
	if(input[i]>(1769+offset1))
{
	if(input[i]==(1771+offset1))
{
	vR[i] = 1771;
}
else
{
	vR[i] = 1770;
}

}
else
{
	if(input[i]==(1769+offset1))
{
	vR[i] = 1769;
}
else
{
	vR[i] = 1768;
}

}

}

}
else
{
	if(input[i]>(1763+offset1))
{
	if(input[i]>(1765+offset1))
{
	if(input[i]==(1767+offset1))
{
	vR[i] = 1767;
}
else
{
	vR[i] = 1766;
}

}
else
{
	if(input[i]==(1765+offset1))
{
	vR[i] = 1765;
}
else
{
	vR[i] = 1764;
}

}

}
else
{
	if(input[i]>(1761+offset1))
{
	if(input[i]==(1763+offset1))
{
	vR[i] = 1763;
}
else
{
	vR[i] = 1762;
}

}
else
{
	if(input[i]==(1761+offset1))
{
	vR[i] = 1761;
}
else
{
	vR[i] = 1760;
}

}

}

}

}

}
else
{
	if(input[i]>(1743+offset1))
{
	if(input[i]>(1751+offset1))
{
	if(input[i]>(1755+offset1))
{
	if(input[i]>(1757+offset1))
{
	if(input[i]==(1759+offset1))
{
	vR[i] = 1759;
}
else
{
	vR[i] = 1758;
}

}
else
{
	if(input[i]==(1757+offset1))
{
	vR[i] = 1757;
}
else
{
	vR[i] = 1756;
}

}

}
else
{
	if(input[i]>(1753+offset1))
{
	if(input[i]==(1755+offset1))
{
	vR[i] = 1755;
}
else
{
	vR[i] = 1754;
}

}
else
{
	if(input[i]==(1753+offset1))
{
	vR[i] = 1753;
}
else
{
	vR[i] = 1752;
}

}

}

}
else
{
	if(input[i]>(1747+offset1))
{
	if(input[i]>(1749+offset1))
{
	if(input[i]==(1751+offset1))
{
	vR[i] = 1751;
}
else
{
	vR[i] = 1750;
}

}
else
{
	if(input[i]==(1749+offset1))
{
	vR[i] = 1749;
}
else
{
	vR[i] = 1748;
}

}

}
else
{
	if(input[i]>(1745+offset1))
{
	if(input[i]==(1747+offset1))
{
	vR[i] = 1747;
}
else
{
	vR[i] = 1746;
}

}
else
{
	if(input[i]==(1745+offset1))
{
	vR[i] = 1745;
}
else
{
	vR[i] = 1744;
}

}

}

}

}
else
{
	if(input[i]>(1735+offset1))
{
	if(input[i]>(1739+offset1))
{
	if(input[i]>(1741+offset1))
{
	if(input[i]==(1743+offset1))
{
	vR[i] = 1743;
}
else
{
	vR[i] = 1742;
}

}
else
{
	if(input[i]==(1741+offset1))
{
	vR[i] = 1741;
}
else
{
	vR[i] = 1740;
}

}

}
else
{
	if(input[i]>(1737+offset1))
{
	if(input[i]==(1739+offset1))
{
	vR[i] = 1739;
}
else
{
	vR[i] = 1738;
}

}
else
{
	if(input[i]==(1737+offset1))
{
	vR[i] = 1737;
}
else
{
	vR[i] = 1736;
}

}

}

}
else
{
	if(input[i]>(1731+offset1))
{
	if(input[i]>(1733+offset1))
{
	if(input[i]==(1735+offset1))
{
	vR[i] = 1735;
}
else
{
	vR[i] = 1734;
}

}
else
{
	if(input[i]==(1733+offset1))
{
	vR[i] = 1733;
}
else
{
	vR[i] = 1732;
}

}

}
else
{
	if(input[i]>(1729+offset1))
{
	if(input[i]==(1731+offset1))
{
	vR[i] = 1731;
}
else
{
	vR[i] = 1730;
}

}
else
{
	if(input[i]==(1729+offset1))
{
	vR[i] = 1729;
}
else
{
	vR[i] = 1728;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1695+offset1))
{
	if(input[i]>(1711+offset1))
{
	if(input[i]>(1719+offset1))
{
	if(input[i]>(1723+offset1))
{
	if(input[i]>(1725+offset1))
{
	if(input[i]==(1727+offset1))
{
	vR[i] = 1727;
}
else
{
	vR[i] = 1726;
}

}
else
{
	if(input[i]==(1725+offset1))
{
	vR[i] = 1725;
}
else
{
	vR[i] = 1724;
}

}

}
else
{
	if(input[i]>(1721+offset1))
{
	if(input[i]==(1723+offset1))
{
	vR[i] = 1723;
}
else
{
	vR[i] = 1722;
}

}
else
{
	if(input[i]==(1721+offset1))
{
	vR[i] = 1721;
}
else
{
	vR[i] = 1720;
}

}

}

}
else
{
	if(input[i]>(1715+offset1))
{
	if(input[i]>(1717+offset1))
{
	if(input[i]==(1719+offset1))
{
	vR[i] = 1719;
}
else
{
	vR[i] = 1718;
}

}
else
{
	if(input[i]==(1717+offset1))
{
	vR[i] = 1717;
}
else
{
	vR[i] = 1716;
}

}

}
else
{
	if(input[i]>(1713+offset1))
{
	if(input[i]==(1715+offset1))
{
	vR[i] = 1715;
}
else
{
	vR[i] = 1714;
}

}
else
{
	if(input[i]==(1713+offset1))
{
	vR[i] = 1713;
}
else
{
	vR[i] = 1712;
}

}

}

}

}
else
{
	if(input[i]>(1703+offset1))
{
	if(input[i]>(1707+offset1))
{
	if(input[i]>(1709+offset1))
{
	if(input[i]==(1711+offset1))
{
	vR[i] = 1711;
}
else
{
	vR[i] = 1710;
}

}
else
{
	if(input[i]==(1709+offset1))
{
	vR[i] = 1709;
}
else
{
	vR[i] = 1708;
}

}

}
else
{
	if(input[i]>(1705+offset1))
{
	if(input[i]==(1707+offset1))
{
	vR[i] = 1707;
}
else
{
	vR[i] = 1706;
}

}
else
{
	if(input[i]==(1705+offset1))
{
	vR[i] = 1705;
}
else
{
	vR[i] = 1704;
}

}

}

}
else
{
	if(input[i]>(1699+offset1))
{
	if(input[i]>(1701+offset1))
{
	if(input[i]==(1703+offset1))
{
	vR[i] = 1703;
}
else
{
	vR[i] = 1702;
}

}
else
{
	if(input[i]==(1701+offset1))
{
	vR[i] = 1701;
}
else
{
	vR[i] = 1700;
}

}

}
else
{
	if(input[i]>(1697+offset1))
{
	if(input[i]==(1699+offset1))
{
	vR[i] = 1699;
}
else
{
	vR[i] = 1698;
}

}
else
{
	if(input[i]==(1697+offset1))
{
	vR[i] = 1697;
}
else
{
	vR[i] = 1696;
}

}

}

}

}

}
else
{
	if(input[i]>(1679+offset1))
{
	if(input[i]>(1687+offset1))
{
	if(input[i]>(1691+offset1))
{
	if(input[i]>(1693+offset1))
{
	if(input[i]==(1695+offset1))
{
	vR[i] = 1695;
}
else
{
	vR[i] = 1694;
}

}
else
{
	if(input[i]==(1693+offset1))
{
	vR[i] = 1693;
}
else
{
	vR[i] = 1692;
}

}

}
else
{
	if(input[i]>(1689+offset1))
{
	if(input[i]==(1691+offset1))
{
	vR[i] = 1691;
}
else
{
	vR[i] = 1690;
}

}
else
{
	if(input[i]==(1689+offset1))
{
	vR[i] = 1689;
}
else
{
	vR[i] = 1688;
}

}

}

}
else
{
	if(input[i]>(1683+offset1))
{
	if(input[i]>(1685+offset1))
{
	if(input[i]==(1687+offset1))
{
	vR[i] = 1687;
}
else
{
	vR[i] = 1686;
}

}
else
{
	if(input[i]==(1685+offset1))
{
	vR[i] = 1685;
}
else
{
	vR[i] = 1684;
}

}

}
else
{
	if(input[i]>(1681+offset1))
{
	if(input[i]==(1683+offset1))
{
	vR[i] = 1683;
}
else
{
	vR[i] = 1682;
}

}
else
{
	if(input[i]==(1681+offset1))
{
	vR[i] = 1681;
}
else
{
	vR[i] = 1680;
}

}

}

}

}
else
{
	if(input[i]>(1671+offset1))
{
	if(input[i]>(1675+offset1))
{
	if(input[i]>(1677+offset1))
{
	if(input[i]==(1679+offset1))
{
	vR[i] = 1679;
}
else
{
	vR[i] = 1678;
}

}
else
{
	if(input[i]==(1677+offset1))
{
	vR[i] = 1677;
}
else
{
	vR[i] = 1676;
}

}

}
else
{
	if(input[i]>(1673+offset1))
{
	if(input[i]==(1675+offset1))
{
	vR[i] = 1675;
}
else
{
	vR[i] = 1674;
}

}
else
{
	if(input[i]==(1673+offset1))
{
	vR[i] = 1673;
}
else
{
	vR[i] = 1672;
}

}

}

}
else
{
	if(input[i]>(1667+offset1))
{
	if(input[i]>(1669+offset1))
{
	if(input[i]==(1671+offset1))
{
	vR[i] = 1671;
}
else
{
	vR[i] = 1670;
}

}
else
{
	if(input[i]==(1669+offset1))
{
	vR[i] = 1669;
}
else
{
	vR[i] = 1668;
}

}

}
else
{
	if(input[i]>(1665+offset1))
{
	if(input[i]==(1667+offset1))
{
	vR[i] = 1667;
}
else
{
	vR[i] = 1666;
}

}
else
{
	if(input[i]==(1665+offset1))
{
	vR[i] = 1665;
}
else
{
	vR[i] = 1664;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(1599+offset1))
{
	if(input[i]>(1631+offset1))
{
	if(input[i]>(1647+offset1))
{
	if(input[i]>(1655+offset1))
{
	if(input[i]>(1659+offset1))
{
	if(input[i]>(1661+offset1))
{
	if(input[i]==(1663+offset1))
{
	vR[i] = 1663;
}
else
{
	vR[i] = 1662;
}

}
else
{
	if(input[i]==(1661+offset1))
{
	vR[i] = 1661;
}
else
{
	vR[i] = 1660;
}

}

}
else
{
	if(input[i]>(1657+offset1))
{
	if(input[i]==(1659+offset1))
{
	vR[i] = 1659;
}
else
{
	vR[i] = 1658;
}

}
else
{
	if(input[i]==(1657+offset1))
{
	vR[i] = 1657;
}
else
{
	vR[i] = 1656;
}

}

}

}
else
{
	if(input[i]>(1651+offset1))
{
	if(input[i]>(1653+offset1))
{
	if(input[i]==(1655+offset1))
{
	vR[i] = 1655;
}
else
{
	vR[i] = 1654;
}

}
else
{
	if(input[i]==(1653+offset1))
{
	vR[i] = 1653;
}
else
{
	vR[i] = 1652;
}

}

}
else
{
	if(input[i]>(1649+offset1))
{
	if(input[i]==(1651+offset1))
{
	vR[i] = 1651;
}
else
{
	vR[i] = 1650;
}

}
else
{
	if(input[i]==(1649+offset1))
{
	vR[i] = 1649;
}
else
{
	vR[i] = 1648;
}

}

}

}

}
else
{
	if(input[i]>(1639+offset1))
{
	if(input[i]>(1643+offset1))
{
	if(input[i]>(1645+offset1))
{
	if(input[i]==(1647+offset1))
{
	vR[i] = 1647;
}
else
{
	vR[i] = 1646;
}

}
else
{
	if(input[i]==(1645+offset1))
{
	vR[i] = 1645;
}
else
{
	vR[i] = 1644;
}

}

}
else
{
	if(input[i]>(1641+offset1))
{
	if(input[i]==(1643+offset1))
{
	vR[i] = 1643;
}
else
{
	vR[i] = 1642;
}

}
else
{
	if(input[i]==(1641+offset1))
{
	vR[i] = 1641;
}
else
{
	vR[i] = 1640;
}

}

}

}
else
{
	if(input[i]>(1635+offset1))
{
	if(input[i]>(1637+offset1))
{
	if(input[i]==(1639+offset1))
{
	vR[i] = 1639;
}
else
{
	vR[i] = 1638;
}

}
else
{
	if(input[i]==(1637+offset1))
{
	vR[i] = 1637;
}
else
{
	vR[i] = 1636;
}

}

}
else
{
	if(input[i]>(1633+offset1))
{
	if(input[i]==(1635+offset1))
{
	vR[i] = 1635;
}
else
{
	vR[i] = 1634;
}

}
else
{
	if(input[i]==(1633+offset1))
{
	vR[i] = 1633;
}
else
{
	vR[i] = 1632;
}

}

}

}

}

}
else
{
	if(input[i]>(1615+offset1))
{
	if(input[i]>(1623+offset1))
{
	if(input[i]>(1627+offset1))
{
	if(input[i]>(1629+offset1))
{
	if(input[i]==(1631+offset1))
{
	vR[i] = 1631;
}
else
{
	vR[i] = 1630;
}

}
else
{
	if(input[i]==(1629+offset1))
{
	vR[i] = 1629;
}
else
{
	vR[i] = 1628;
}

}

}
else
{
	if(input[i]>(1625+offset1))
{
	if(input[i]==(1627+offset1))
{
	vR[i] = 1627;
}
else
{
	vR[i] = 1626;
}

}
else
{
	if(input[i]==(1625+offset1))
{
	vR[i] = 1625;
}
else
{
	vR[i] = 1624;
}

}

}

}
else
{
	if(input[i]>(1619+offset1))
{
	if(input[i]>(1621+offset1))
{
	if(input[i]==(1623+offset1))
{
	vR[i] = 1623;
}
else
{
	vR[i] = 1622;
}

}
else
{
	if(input[i]==(1621+offset1))
{
	vR[i] = 1621;
}
else
{
	vR[i] = 1620;
}

}

}
else
{
	if(input[i]>(1617+offset1))
{
	if(input[i]==(1619+offset1))
{
	vR[i] = 1619;
}
else
{
	vR[i] = 1618;
}

}
else
{
	if(input[i]==(1617+offset1))
{
	vR[i] = 1617;
}
else
{
	vR[i] = 1616;
}

}

}

}

}
else
{
	if(input[i]>(1607+offset1))
{
	if(input[i]>(1611+offset1))
{
	if(input[i]>(1613+offset1))
{
	if(input[i]==(1615+offset1))
{
	vR[i] = 1615;
}
else
{
	vR[i] = 1614;
}

}
else
{
	if(input[i]==(1613+offset1))
{
	vR[i] = 1613;
}
else
{
	vR[i] = 1612;
}

}

}
else
{
	if(input[i]>(1609+offset1))
{
	if(input[i]==(1611+offset1))
{
	vR[i] = 1611;
}
else
{
	vR[i] = 1610;
}

}
else
{
	if(input[i]==(1609+offset1))
{
	vR[i] = 1609;
}
else
{
	vR[i] = 1608;
}

}

}

}
else
{
	if(input[i]>(1603+offset1))
{
	if(input[i]>(1605+offset1))
{
	if(input[i]==(1607+offset1))
{
	vR[i] = 1607;
}
else
{
	vR[i] = 1606;
}

}
else
{
	if(input[i]==(1605+offset1))
{
	vR[i] = 1605;
}
else
{
	vR[i] = 1604;
}

}

}
else
{
	if(input[i]>(1601+offset1))
{
	if(input[i]==(1603+offset1))
{
	vR[i] = 1603;
}
else
{
	vR[i] = 1602;
}

}
else
{
	if(input[i]==(1601+offset1))
{
	vR[i] = 1601;
}
else
{
	vR[i] = 1600;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1567+offset1))
{
	if(input[i]>(1583+offset1))
{
	if(input[i]>(1591+offset1))
{
	if(input[i]>(1595+offset1))
{
	if(input[i]>(1597+offset1))
{
	if(input[i]==(1599+offset1))
{
	vR[i] = 1599;
}
else
{
	vR[i] = 1598;
}

}
else
{
	if(input[i]==(1597+offset1))
{
	vR[i] = 1597;
}
else
{
	vR[i] = 1596;
}

}

}
else
{
	if(input[i]>(1593+offset1))
{
	if(input[i]==(1595+offset1))
{
	vR[i] = 1595;
}
else
{
	vR[i] = 1594;
}

}
else
{
	if(input[i]==(1593+offset1))
{
	vR[i] = 1593;
}
else
{
	vR[i] = 1592;
}

}

}

}
else
{
	if(input[i]>(1587+offset1))
{
	if(input[i]>(1589+offset1))
{
	if(input[i]==(1591+offset1))
{
	vR[i] = 1591;
}
else
{
	vR[i] = 1590;
}

}
else
{
	if(input[i]==(1589+offset1))
{
	vR[i] = 1589;
}
else
{
	vR[i] = 1588;
}

}

}
else
{
	if(input[i]>(1585+offset1))
{
	if(input[i]==(1587+offset1))
{
	vR[i] = 1587;
}
else
{
	vR[i] = 1586;
}

}
else
{
	if(input[i]==(1585+offset1))
{
	vR[i] = 1585;
}
else
{
	vR[i] = 1584;
}

}

}

}

}
else
{
	if(input[i]>(1575+offset1))
{
	if(input[i]>(1579+offset1))
{
	if(input[i]>(1581+offset1))
{
	if(input[i]==(1583+offset1))
{
	vR[i] = 1583;
}
else
{
	vR[i] = 1582;
}

}
else
{
	if(input[i]==(1581+offset1))
{
	vR[i] = 1581;
}
else
{
	vR[i] = 1580;
}

}

}
else
{
	if(input[i]>(1577+offset1))
{
	if(input[i]==(1579+offset1))
{
	vR[i] = 1579;
}
else
{
	vR[i] = 1578;
}

}
else
{
	if(input[i]==(1577+offset1))
{
	vR[i] = 1577;
}
else
{
	vR[i] = 1576;
}

}

}

}
else
{
	if(input[i]>(1571+offset1))
{
	if(input[i]>(1573+offset1))
{
	if(input[i]==(1575+offset1))
{
	vR[i] = 1575;
}
else
{
	vR[i] = 1574;
}

}
else
{
	if(input[i]==(1573+offset1))
{
	vR[i] = 1573;
}
else
{
	vR[i] = 1572;
}

}

}
else
{
	if(input[i]>(1569+offset1))
{
	if(input[i]==(1571+offset1))
{
	vR[i] = 1571;
}
else
{
	vR[i] = 1570;
}

}
else
{
	if(input[i]==(1569+offset1))
{
	vR[i] = 1569;
}
else
{
	vR[i] = 1568;
}

}

}

}

}

}
else
{
	if(input[i]>(1551+offset1))
{
	if(input[i]>(1559+offset1))
{
	if(input[i]>(1563+offset1))
{
	if(input[i]>(1565+offset1))
{
	if(input[i]==(1567+offset1))
{
	vR[i] = 1567;
}
else
{
	vR[i] = 1566;
}

}
else
{
	if(input[i]==(1565+offset1))
{
	vR[i] = 1565;
}
else
{
	vR[i] = 1564;
}

}

}
else
{
	if(input[i]>(1561+offset1))
{
	if(input[i]==(1563+offset1))
{
	vR[i] = 1563;
}
else
{
	vR[i] = 1562;
}

}
else
{
	if(input[i]==(1561+offset1))
{
	vR[i] = 1561;
}
else
{
	vR[i] = 1560;
}

}

}

}
else
{
	if(input[i]>(1555+offset1))
{
	if(input[i]>(1557+offset1))
{
	if(input[i]==(1559+offset1))
{
	vR[i] = 1559;
}
else
{
	vR[i] = 1558;
}

}
else
{
	if(input[i]==(1557+offset1))
{
	vR[i] = 1557;
}
else
{
	vR[i] = 1556;
}

}

}
else
{
	if(input[i]>(1553+offset1))
{
	if(input[i]==(1555+offset1))
{
	vR[i] = 1555;
}
else
{
	vR[i] = 1554;
}

}
else
{
	if(input[i]==(1553+offset1))
{
	vR[i] = 1553;
}
else
{
	vR[i] = 1552;
}

}

}

}

}
else
{
	if(input[i]>(1543+offset1))
{
	if(input[i]>(1547+offset1))
{
	if(input[i]>(1549+offset1))
{
	if(input[i]==(1551+offset1))
{
	vR[i] = 1551;
}
else
{
	vR[i] = 1550;
}

}
else
{
	if(input[i]==(1549+offset1))
{
	vR[i] = 1549;
}
else
{
	vR[i] = 1548;
}

}

}
else
{
	if(input[i]>(1545+offset1))
{
	if(input[i]==(1547+offset1))
{
	vR[i] = 1547;
}
else
{
	vR[i] = 1546;
}

}
else
{
	if(input[i]==(1545+offset1))
{
	vR[i] = 1545;
}
else
{
	vR[i] = 1544;
}

}

}

}
else
{
	if(input[i]>(1539+offset1))
{
	if(input[i]>(1541+offset1))
{
	if(input[i]==(1543+offset1))
{
	vR[i] = 1543;
}
else
{
	vR[i] = 1542;
}

}
else
{
	if(input[i]==(1541+offset1))
{
	vR[i] = 1541;
}
else
{
	vR[i] = 1540;
}

}

}
else
{
	if(input[i]>(1537+offset1))
{
	if(input[i]==(1539+offset1))
{
	vR[i] = 1539;
}
else
{
	vR[i] = 1538;
}

}
else
{
	if(input[i]==(1537+offset1))
{
	vR[i] = 1537;
}
else
{
	vR[i] = 1536;
}

}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(1279+offset1))
{
	if(input[i]>(1407+offset1))
{
	if(input[i]>(1471+offset1))
{
	if(input[i]>(1503+offset1))
{
	if(input[i]>(1519+offset1))
{
	if(input[i]>(1527+offset1))
{
	if(input[i]>(1531+offset1))
{
	if(input[i]>(1533+offset1))
{
	if(input[i]==(1535+offset1))
{
	vR[i] = 1535;
}
else
{
	vR[i] = 1534;
}

}
else
{
	if(input[i]==(1533+offset1))
{
	vR[i] = 1533;
}
else
{
	vR[i] = 1532;
}

}

}
else
{
	if(input[i]>(1529+offset1))
{
	if(input[i]==(1531+offset1))
{
	vR[i] = 1531;
}
else
{
	vR[i] = 1530;
}

}
else
{
	if(input[i]==(1529+offset1))
{
	vR[i] = 1529;
}
else
{
	vR[i] = 1528;
}

}

}

}
else
{
	if(input[i]>(1523+offset1))
{
	if(input[i]>(1525+offset1))
{
	if(input[i]==(1527+offset1))
{
	vR[i] = 1527;
}
else
{
	vR[i] = 1526;
}

}
else
{
	if(input[i]==(1525+offset1))
{
	vR[i] = 1525;
}
else
{
	vR[i] = 1524;
}

}

}
else
{
	if(input[i]>(1521+offset1))
{
	if(input[i]==(1523+offset1))
{
	vR[i] = 1523;
}
else
{
	vR[i] = 1522;
}

}
else
{
	if(input[i]==(1521+offset1))
{
	vR[i] = 1521;
}
else
{
	vR[i] = 1520;
}

}

}

}

}
else
{
	if(input[i]>(1511+offset1))
{
	if(input[i]>(1515+offset1))
{
	if(input[i]>(1517+offset1))
{
	if(input[i]==(1519+offset1))
{
	vR[i] = 1519;
}
else
{
	vR[i] = 1518;
}

}
else
{
	if(input[i]==(1517+offset1))
{
	vR[i] = 1517;
}
else
{
	vR[i] = 1516;
}

}

}
else
{
	if(input[i]>(1513+offset1))
{
	if(input[i]==(1515+offset1))
{
	vR[i] = 1515;
}
else
{
	vR[i] = 1514;
}

}
else
{
	if(input[i]==(1513+offset1))
{
	vR[i] = 1513;
}
else
{
	vR[i] = 1512;
}

}

}

}
else
{
	if(input[i]>(1507+offset1))
{
	if(input[i]>(1509+offset1))
{
	if(input[i]==(1511+offset1))
{
	vR[i] = 1511;
}
else
{
	vR[i] = 1510;
}

}
else
{
	if(input[i]==(1509+offset1))
{
	vR[i] = 1509;
}
else
{
	vR[i] = 1508;
}

}

}
else
{
	if(input[i]>(1505+offset1))
{
	if(input[i]==(1507+offset1))
{
	vR[i] = 1507;
}
else
{
	vR[i] = 1506;
}

}
else
{
	if(input[i]==(1505+offset1))
{
	vR[i] = 1505;
}
else
{
	vR[i] = 1504;
}

}

}

}

}

}
else
{
	if(input[i]>(1487+offset1))
{
	if(input[i]>(1495+offset1))
{
	if(input[i]>(1499+offset1))
{
	if(input[i]>(1501+offset1))
{
	if(input[i]==(1503+offset1))
{
	vR[i] = 1503;
}
else
{
	vR[i] = 1502;
}

}
else
{
	if(input[i]==(1501+offset1))
{
	vR[i] = 1501;
}
else
{
	vR[i] = 1500;
}

}

}
else
{
	if(input[i]>(1497+offset1))
{
	if(input[i]==(1499+offset1))
{
	vR[i] = 1499;
}
else
{
	vR[i] = 1498;
}

}
else
{
	if(input[i]==(1497+offset1))
{
	vR[i] = 1497;
}
else
{
	vR[i] = 1496;
}

}

}

}
else
{
	if(input[i]>(1491+offset1))
{
	if(input[i]>(1493+offset1))
{
	if(input[i]==(1495+offset1))
{
	vR[i] = 1495;
}
else
{
	vR[i] = 1494;
}

}
else
{
	if(input[i]==(1493+offset1))
{
	vR[i] = 1493;
}
else
{
	vR[i] = 1492;
}

}

}
else
{
	if(input[i]>(1489+offset1))
{
	if(input[i]==(1491+offset1))
{
	vR[i] = 1491;
}
else
{
	vR[i] = 1490;
}

}
else
{
	if(input[i]==(1489+offset1))
{
	vR[i] = 1489;
}
else
{
	vR[i] = 1488;
}

}

}

}

}
else
{
	if(input[i]>(1479+offset1))
{
	if(input[i]>(1483+offset1))
{
	if(input[i]>(1485+offset1))
{
	if(input[i]==(1487+offset1))
{
	vR[i] = 1487;
}
else
{
	vR[i] = 1486;
}

}
else
{
	if(input[i]==(1485+offset1))
{
	vR[i] = 1485;
}
else
{
	vR[i] = 1484;
}

}

}
else
{
	if(input[i]>(1481+offset1))
{
	if(input[i]==(1483+offset1))
{
	vR[i] = 1483;
}
else
{
	vR[i] = 1482;
}

}
else
{
	if(input[i]==(1481+offset1))
{
	vR[i] = 1481;
}
else
{
	vR[i] = 1480;
}

}

}

}
else
{
	if(input[i]>(1475+offset1))
{
	if(input[i]>(1477+offset1))
{
	if(input[i]==(1479+offset1))
{
	vR[i] = 1479;
}
else
{
	vR[i] = 1478;
}

}
else
{
	if(input[i]==(1477+offset1))
{
	vR[i] = 1477;
}
else
{
	vR[i] = 1476;
}

}

}
else
{
	if(input[i]>(1473+offset1))
{
	if(input[i]==(1475+offset1))
{
	vR[i] = 1475;
}
else
{
	vR[i] = 1474;
}

}
else
{
	if(input[i]==(1473+offset1))
{
	vR[i] = 1473;
}
else
{
	vR[i] = 1472;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1439+offset1))
{
	if(input[i]>(1455+offset1))
{
	if(input[i]>(1463+offset1))
{
	if(input[i]>(1467+offset1))
{
	if(input[i]>(1469+offset1))
{
	if(input[i]==(1471+offset1))
{
	vR[i] = 1471;
}
else
{
	vR[i] = 1470;
}

}
else
{
	if(input[i]==(1469+offset1))
{
	vR[i] = 1469;
}
else
{
	vR[i] = 1468;
}

}

}
else
{
	if(input[i]>(1465+offset1))
{
	if(input[i]==(1467+offset1))
{
	vR[i] = 1467;
}
else
{
	vR[i] = 1466;
}

}
else
{
	if(input[i]==(1465+offset1))
{
	vR[i] = 1465;
}
else
{
	vR[i] = 1464;
}

}

}

}
else
{
	if(input[i]>(1459+offset1))
{
	if(input[i]>(1461+offset1))
{
	if(input[i]==(1463+offset1))
{
	vR[i] = 1463;
}
else
{
	vR[i] = 1462;
}

}
else
{
	if(input[i]==(1461+offset1))
{
	vR[i] = 1461;
}
else
{
	vR[i] = 1460;
}

}

}
else
{
	if(input[i]>(1457+offset1))
{
	if(input[i]==(1459+offset1))
{
	vR[i] = 1459;
}
else
{
	vR[i] = 1458;
}

}
else
{
	if(input[i]==(1457+offset1))
{
	vR[i] = 1457;
}
else
{
	vR[i] = 1456;
}

}

}

}

}
else
{
	if(input[i]>(1447+offset1))
{
	if(input[i]>(1451+offset1))
{
	if(input[i]>(1453+offset1))
{
	if(input[i]==(1455+offset1))
{
	vR[i] = 1455;
}
else
{
	vR[i] = 1454;
}

}
else
{
	if(input[i]==(1453+offset1))
{
	vR[i] = 1453;
}
else
{
	vR[i] = 1452;
}

}

}
else
{
	if(input[i]>(1449+offset1))
{
	if(input[i]==(1451+offset1))
{
	vR[i] = 1451;
}
else
{
	vR[i] = 1450;
}

}
else
{
	if(input[i]==(1449+offset1))
{
	vR[i] = 1449;
}
else
{
	vR[i] = 1448;
}

}

}

}
else
{
	if(input[i]>(1443+offset1))
{
	if(input[i]>(1445+offset1))
{
	if(input[i]==(1447+offset1))
{
	vR[i] = 1447;
}
else
{
	vR[i] = 1446;
}

}
else
{
	if(input[i]==(1445+offset1))
{
	vR[i] = 1445;
}
else
{
	vR[i] = 1444;
}

}

}
else
{
	if(input[i]>(1441+offset1))
{
	if(input[i]==(1443+offset1))
{
	vR[i] = 1443;
}
else
{
	vR[i] = 1442;
}

}
else
{
	if(input[i]==(1441+offset1))
{
	vR[i] = 1441;
}
else
{
	vR[i] = 1440;
}

}

}

}

}

}
else
{
	if(input[i]>(1423+offset1))
{
	if(input[i]>(1431+offset1))
{
	if(input[i]>(1435+offset1))
{
	if(input[i]>(1437+offset1))
{
	if(input[i]==(1439+offset1))
{
	vR[i] = 1439;
}
else
{
	vR[i] = 1438;
}

}
else
{
	if(input[i]==(1437+offset1))
{
	vR[i] = 1437;
}
else
{
	vR[i] = 1436;
}

}

}
else
{
	if(input[i]>(1433+offset1))
{
	if(input[i]==(1435+offset1))
{
	vR[i] = 1435;
}
else
{
	vR[i] = 1434;
}

}
else
{
	if(input[i]==(1433+offset1))
{
	vR[i] = 1433;
}
else
{
	vR[i] = 1432;
}

}

}

}
else
{
	if(input[i]>(1427+offset1))
{
	if(input[i]>(1429+offset1))
{
	if(input[i]==(1431+offset1))
{
	vR[i] = 1431;
}
else
{
	vR[i] = 1430;
}

}
else
{
	if(input[i]==(1429+offset1))
{
	vR[i] = 1429;
}
else
{
	vR[i] = 1428;
}

}

}
else
{
	if(input[i]>(1425+offset1))
{
	if(input[i]==(1427+offset1))
{
	vR[i] = 1427;
}
else
{
	vR[i] = 1426;
}

}
else
{
	if(input[i]==(1425+offset1))
{
	vR[i] = 1425;
}
else
{
	vR[i] = 1424;
}

}

}

}

}
else
{
	if(input[i]>(1415+offset1))
{
	if(input[i]>(1419+offset1))
{
	if(input[i]>(1421+offset1))
{
	if(input[i]==(1423+offset1))
{
	vR[i] = 1423;
}
else
{
	vR[i] = 1422;
}

}
else
{
	if(input[i]==(1421+offset1))
{
	vR[i] = 1421;
}
else
{
	vR[i] = 1420;
}

}

}
else
{
	if(input[i]>(1417+offset1))
{
	if(input[i]==(1419+offset1))
{
	vR[i] = 1419;
}
else
{
	vR[i] = 1418;
}

}
else
{
	if(input[i]==(1417+offset1))
{
	vR[i] = 1417;
}
else
{
	vR[i] = 1416;
}

}

}

}
else
{
	if(input[i]>(1411+offset1))
{
	if(input[i]>(1413+offset1))
{
	if(input[i]==(1415+offset1))
{
	vR[i] = 1415;
}
else
{
	vR[i] = 1414;
}

}
else
{
	if(input[i]==(1413+offset1))
{
	vR[i] = 1413;
}
else
{
	vR[i] = 1412;
}

}

}
else
{
	if(input[i]>(1409+offset1))
{
	if(input[i]==(1411+offset1))
{
	vR[i] = 1411;
}
else
{
	vR[i] = 1410;
}

}
else
{
	if(input[i]==(1409+offset1))
{
	vR[i] = 1409;
}
else
{
	vR[i] = 1408;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(1343+offset1))
{
	if(input[i]>(1375+offset1))
{
	if(input[i]>(1391+offset1))
{
	if(input[i]>(1399+offset1))
{
	if(input[i]>(1403+offset1))
{
	if(input[i]>(1405+offset1))
{
	if(input[i]==(1407+offset1))
{
	vR[i] = 1407;
}
else
{
	vR[i] = 1406;
}

}
else
{
	if(input[i]==(1405+offset1))
{
	vR[i] = 1405;
}
else
{
	vR[i] = 1404;
}

}

}
else
{
	if(input[i]>(1401+offset1))
{
	if(input[i]==(1403+offset1))
{
	vR[i] = 1403;
}
else
{
	vR[i] = 1402;
}

}
else
{
	if(input[i]==(1401+offset1))
{
	vR[i] = 1401;
}
else
{
	vR[i] = 1400;
}

}

}

}
else
{
	if(input[i]>(1395+offset1))
{
	if(input[i]>(1397+offset1))
{
	if(input[i]==(1399+offset1))
{
	vR[i] = 1399;
}
else
{
	vR[i] = 1398;
}

}
else
{
	if(input[i]==(1397+offset1))
{
	vR[i] = 1397;
}
else
{
	vR[i] = 1396;
}

}

}
else
{
	if(input[i]>(1393+offset1))
{
	if(input[i]==(1395+offset1))
{
	vR[i] = 1395;
}
else
{
	vR[i] = 1394;
}

}
else
{
	if(input[i]==(1393+offset1))
{
	vR[i] = 1393;
}
else
{
	vR[i] = 1392;
}

}

}

}

}
else
{
	if(input[i]>(1383+offset1))
{
	if(input[i]>(1387+offset1))
{
	if(input[i]>(1389+offset1))
{
	if(input[i]==(1391+offset1))
{
	vR[i] = 1391;
}
else
{
	vR[i] = 1390;
}

}
else
{
	if(input[i]==(1389+offset1))
{
	vR[i] = 1389;
}
else
{
	vR[i] = 1388;
}

}

}
else
{
	if(input[i]>(1385+offset1))
{
	if(input[i]==(1387+offset1))
{
	vR[i] = 1387;
}
else
{
	vR[i] = 1386;
}

}
else
{
	if(input[i]==(1385+offset1))
{
	vR[i] = 1385;
}
else
{
	vR[i] = 1384;
}

}

}

}
else
{
	if(input[i]>(1379+offset1))
{
	if(input[i]>(1381+offset1))
{
	if(input[i]==(1383+offset1))
{
	vR[i] = 1383;
}
else
{
	vR[i] = 1382;
}

}
else
{
	if(input[i]==(1381+offset1))
{
	vR[i] = 1381;
}
else
{
	vR[i] = 1380;
}

}

}
else
{
	if(input[i]>(1377+offset1))
{
	if(input[i]==(1379+offset1))
{
	vR[i] = 1379;
}
else
{
	vR[i] = 1378;
}

}
else
{
	if(input[i]==(1377+offset1))
{
	vR[i] = 1377;
}
else
{
	vR[i] = 1376;
}

}

}

}

}

}
else
{
	if(input[i]>(1359+offset1))
{
	if(input[i]>(1367+offset1))
{
	if(input[i]>(1371+offset1))
{
	if(input[i]>(1373+offset1))
{
	if(input[i]==(1375+offset1))
{
	vR[i] = 1375;
}
else
{
	vR[i] = 1374;
}

}
else
{
	if(input[i]==(1373+offset1))
{
	vR[i] = 1373;
}
else
{
	vR[i] = 1372;
}

}

}
else
{
	if(input[i]>(1369+offset1))
{
	if(input[i]==(1371+offset1))
{
	vR[i] = 1371;
}
else
{
	vR[i] = 1370;
}

}
else
{
	if(input[i]==(1369+offset1))
{
	vR[i] = 1369;
}
else
{
	vR[i] = 1368;
}

}

}

}
else
{
	if(input[i]>(1363+offset1))
{
	if(input[i]>(1365+offset1))
{
	if(input[i]==(1367+offset1))
{
	vR[i] = 1367;
}
else
{
	vR[i] = 1366;
}

}
else
{
	if(input[i]==(1365+offset1))
{
	vR[i] = 1365;
}
else
{
	vR[i] = 1364;
}

}

}
else
{
	if(input[i]>(1361+offset1))
{
	if(input[i]==(1363+offset1))
{
	vR[i] = 1363;
}
else
{
	vR[i] = 1362;
}

}
else
{
	if(input[i]==(1361+offset1))
{
	vR[i] = 1361;
}
else
{
	vR[i] = 1360;
}

}

}

}

}
else
{
	if(input[i]>(1351+offset1))
{
	if(input[i]>(1355+offset1))
{
	if(input[i]>(1357+offset1))
{
	if(input[i]==(1359+offset1))
{
	vR[i] = 1359;
}
else
{
	vR[i] = 1358;
}

}
else
{
	if(input[i]==(1357+offset1))
{
	vR[i] = 1357;
}
else
{
	vR[i] = 1356;
}

}

}
else
{
	if(input[i]>(1353+offset1))
{
	if(input[i]==(1355+offset1))
{
	vR[i] = 1355;
}
else
{
	vR[i] = 1354;
}

}
else
{
	if(input[i]==(1353+offset1))
{
	vR[i] = 1353;
}
else
{
	vR[i] = 1352;
}

}

}

}
else
{
	if(input[i]>(1347+offset1))
{
	if(input[i]>(1349+offset1))
{
	if(input[i]==(1351+offset1))
{
	vR[i] = 1351;
}
else
{
	vR[i] = 1350;
}

}
else
{
	if(input[i]==(1349+offset1))
{
	vR[i] = 1349;
}
else
{
	vR[i] = 1348;
}

}

}
else
{
	if(input[i]>(1345+offset1))
{
	if(input[i]==(1347+offset1))
{
	vR[i] = 1347;
}
else
{
	vR[i] = 1346;
}

}
else
{
	if(input[i]==(1345+offset1))
{
	vR[i] = 1345;
}
else
{
	vR[i] = 1344;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1311+offset1))
{
	if(input[i]>(1327+offset1))
{
	if(input[i]>(1335+offset1))
{
	if(input[i]>(1339+offset1))
{
	if(input[i]>(1341+offset1))
{
	if(input[i]==(1343+offset1))
{
	vR[i] = 1343;
}
else
{
	vR[i] = 1342;
}

}
else
{
	if(input[i]==(1341+offset1))
{
	vR[i] = 1341;
}
else
{
	vR[i] = 1340;
}

}

}
else
{
	if(input[i]>(1337+offset1))
{
	if(input[i]==(1339+offset1))
{
	vR[i] = 1339;
}
else
{
	vR[i] = 1338;
}

}
else
{
	if(input[i]==(1337+offset1))
{
	vR[i] = 1337;
}
else
{
	vR[i] = 1336;
}

}

}

}
else
{
	if(input[i]>(1331+offset1))
{
	if(input[i]>(1333+offset1))
{
	if(input[i]==(1335+offset1))
{
	vR[i] = 1335;
}
else
{
	vR[i] = 1334;
}

}
else
{
	if(input[i]==(1333+offset1))
{
	vR[i] = 1333;
}
else
{
	vR[i] = 1332;
}

}

}
else
{
	if(input[i]>(1329+offset1))
{
	if(input[i]==(1331+offset1))
{
	vR[i] = 1331;
}
else
{
	vR[i] = 1330;
}

}
else
{
	if(input[i]==(1329+offset1))
{
	vR[i] = 1329;
}
else
{
	vR[i] = 1328;
}

}

}

}

}
else
{
	if(input[i]>(1319+offset1))
{
	if(input[i]>(1323+offset1))
{
	if(input[i]>(1325+offset1))
{
	if(input[i]==(1327+offset1))
{
	vR[i] = 1327;
}
else
{
	vR[i] = 1326;
}

}
else
{
	if(input[i]==(1325+offset1))
{
	vR[i] = 1325;
}
else
{
	vR[i] = 1324;
}

}

}
else
{
	if(input[i]>(1321+offset1))
{
	if(input[i]==(1323+offset1))
{
	vR[i] = 1323;
}
else
{
	vR[i] = 1322;
}

}
else
{
	if(input[i]==(1321+offset1))
{
	vR[i] = 1321;
}
else
{
	vR[i] = 1320;
}

}

}

}
else
{
	if(input[i]>(1315+offset1))
{
	if(input[i]>(1317+offset1))
{
	if(input[i]==(1319+offset1))
{
	vR[i] = 1319;
}
else
{
	vR[i] = 1318;
}

}
else
{
	if(input[i]==(1317+offset1))
{
	vR[i] = 1317;
}
else
{
	vR[i] = 1316;
}

}

}
else
{
	if(input[i]>(1313+offset1))
{
	if(input[i]==(1315+offset1))
{
	vR[i] = 1315;
}
else
{
	vR[i] = 1314;
}

}
else
{
	if(input[i]==(1313+offset1))
{
	vR[i] = 1313;
}
else
{
	vR[i] = 1312;
}

}

}

}

}

}
else
{
	if(input[i]>(1295+offset1))
{
	if(input[i]>(1303+offset1))
{
	if(input[i]>(1307+offset1))
{
	if(input[i]>(1309+offset1))
{
	if(input[i]==(1311+offset1))
{
	vR[i] = 1311;
}
else
{
	vR[i] = 1310;
}

}
else
{
	if(input[i]==(1309+offset1))
{
	vR[i] = 1309;
}
else
{
	vR[i] = 1308;
}

}

}
else
{
	if(input[i]>(1305+offset1))
{
	if(input[i]==(1307+offset1))
{
	vR[i] = 1307;
}
else
{
	vR[i] = 1306;
}

}
else
{
	if(input[i]==(1305+offset1))
{
	vR[i] = 1305;
}
else
{
	vR[i] = 1304;
}

}

}

}
else
{
	if(input[i]>(1299+offset1))
{
	if(input[i]>(1301+offset1))
{
	if(input[i]==(1303+offset1))
{
	vR[i] = 1303;
}
else
{
	vR[i] = 1302;
}

}
else
{
	if(input[i]==(1301+offset1))
{
	vR[i] = 1301;
}
else
{
	vR[i] = 1300;
}

}

}
else
{
	if(input[i]>(1297+offset1))
{
	if(input[i]==(1299+offset1))
{
	vR[i] = 1299;
}
else
{
	vR[i] = 1298;
}

}
else
{
	if(input[i]==(1297+offset1))
{
	vR[i] = 1297;
}
else
{
	vR[i] = 1296;
}

}

}

}

}
else
{
	if(input[i]>(1287+offset1))
{
	if(input[i]>(1291+offset1))
{
	if(input[i]>(1293+offset1))
{
	if(input[i]==(1295+offset1))
{
	vR[i] = 1295;
}
else
{
	vR[i] = 1294;
}

}
else
{
	if(input[i]==(1293+offset1))
{
	vR[i] = 1293;
}
else
{
	vR[i] = 1292;
}

}

}
else
{
	if(input[i]>(1289+offset1))
{
	if(input[i]==(1291+offset1))
{
	vR[i] = 1291;
}
else
{
	vR[i] = 1290;
}

}
else
{
	if(input[i]==(1289+offset1))
{
	vR[i] = 1289;
}
else
{
	vR[i] = 1288;
}

}

}

}
else
{
	if(input[i]>(1283+offset1))
{
	if(input[i]>(1285+offset1))
{
	if(input[i]==(1287+offset1))
{
	vR[i] = 1287;
}
else
{
	vR[i] = 1286;
}

}
else
{
	if(input[i]==(1285+offset1))
{
	vR[i] = 1285;
}
else
{
	vR[i] = 1284;
}

}

}
else
{
	if(input[i]>(1281+offset1))
{
	if(input[i]==(1283+offset1))
{
	vR[i] = 1283;
}
else
{
	vR[i] = 1282;
}

}
else
{
	if(input[i]==(1281+offset1))
{
	vR[i] = 1281;
}
else
{
	vR[i] = 1280;
}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(1151+offset1))
{
	if(input[i]>(1215+offset1))
{
	if(input[i]>(1247+offset1))
{
	if(input[i]>(1263+offset1))
{
	if(input[i]>(1271+offset1))
{
	if(input[i]>(1275+offset1))
{
	if(input[i]>(1277+offset1))
{
	if(input[i]==(1279+offset1))
{
	vR[i] = 1279;
}
else
{
	vR[i] = 1278;
}

}
else
{
	if(input[i]==(1277+offset1))
{
	vR[i] = 1277;
}
else
{
	vR[i] = 1276;
}

}

}
else
{
	if(input[i]>(1273+offset1))
{
	if(input[i]==(1275+offset1))
{
	vR[i] = 1275;
}
else
{
	vR[i] = 1274;
}

}
else
{
	if(input[i]==(1273+offset1))
{
	vR[i] = 1273;
}
else
{
	vR[i] = 1272;
}

}

}

}
else
{
	if(input[i]>(1267+offset1))
{
	if(input[i]>(1269+offset1))
{
	if(input[i]==(1271+offset1))
{
	vR[i] = 1271;
}
else
{
	vR[i] = 1270;
}

}
else
{
	if(input[i]==(1269+offset1))
{
	vR[i] = 1269;
}
else
{
	vR[i] = 1268;
}

}

}
else
{
	if(input[i]>(1265+offset1))
{
	if(input[i]==(1267+offset1))
{
	vR[i] = 1267;
}
else
{
	vR[i] = 1266;
}

}
else
{
	if(input[i]==(1265+offset1))
{
	vR[i] = 1265;
}
else
{
	vR[i] = 1264;
}

}

}

}

}
else
{
	if(input[i]>(1255+offset1))
{
	if(input[i]>(1259+offset1))
{
	if(input[i]>(1261+offset1))
{
	if(input[i]==(1263+offset1))
{
	vR[i] = 1263;
}
else
{
	vR[i] = 1262;
}

}
else
{
	if(input[i]==(1261+offset1))
{
	vR[i] = 1261;
}
else
{
	vR[i] = 1260;
}

}

}
else
{
	if(input[i]>(1257+offset1))
{
	if(input[i]==(1259+offset1))
{
	vR[i] = 1259;
}
else
{
	vR[i] = 1258;
}

}
else
{
	if(input[i]==(1257+offset1))
{
	vR[i] = 1257;
}
else
{
	vR[i] = 1256;
}

}

}

}
else
{
	if(input[i]>(1251+offset1))
{
	if(input[i]>(1253+offset1))
{
	if(input[i]==(1255+offset1))
{
	vR[i] = 1255;
}
else
{
	vR[i] = 1254;
}

}
else
{
	if(input[i]==(1253+offset1))
{
	vR[i] = 1253;
}
else
{
	vR[i] = 1252;
}

}

}
else
{
	if(input[i]>(1249+offset1))
{
	if(input[i]==(1251+offset1))
{
	vR[i] = 1251;
}
else
{
	vR[i] = 1250;
}

}
else
{
	if(input[i]==(1249+offset1))
{
	vR[i] = 1249;
}
else
{
	vR[i] = 1248;
}

}

}

}

}

}
else
{
	if(input[i]>(1231+offset1))
{
	if(input[i]>(1239+offset1))
{
	if(input[i]>(1243+offset1))
{
	if(input[i]>(1245+offset1))
{
	if(input[i]==(1247+offset1))
{
	vR[i] = 1247;
}
else
{
	vR[i] = 1246;
}

}
else
{
	if(input[i]==(1245+offset1))
{
	vR[i] = 1245;
}
else
{
	vR[i] = 1244;
}

}

}
else
{
	if(input[i]>(1241+offset1))
{
	if(input[i]==(1243+offset1))
{
	vR[i] = 1243;
}
else
{
	vR[i] = 1242;
}

}
else
{
	if(input[i]==(1241+offset1))
{
	vR[i] = 1241;
}
else
{
	vR[i] = 1240;
}

}

}

}
else
{
	if(input[i]>(1235+offset1))
{
	if(input[i]>(1237+offset1))
{
	if(input[i]==(1239+offset1))
{
	vR[i] = 1239;
}
else
{
	vR[i] = 1238;
}

}
else
{
	if(input[i]==(1237+offset1))
{
	vR[i] = 1237;
}
else
{
	vR[i] = 1236;
}

}

}
else
{
	if(input[i]>(1233+offset1))
{
	if(input[i]==(1235+offset1))
{
	vR[i] = 1235;
}
else
{
	vR[i] = 1234;
}

}
else
{
	if(input[i]==(1233+offset1))
{
	vR[i] = 1233;
}
else
{
	vR[i] = 1232;
}

}

}

}

}
else
{
	if(input[i]>(1223+offset1))
{
	if(input[i]>(1227+offset1))
{
	if(input[i]>(1229+offset1))
{
	if(input[i]==(1231+offset1))
{
	vR[i] = 1231;
}
else
{
	vR[i] = 1230;
}

}
else
{
	if(input[i]==(1229+offset1))
{
	vR[i] = 1229;
}
else
{
	vR[i] = 1228;
}

}

}
else
{
	if(input[i]>(1225+offset1))
{
	if(input[i]==(1227+offset1))
{
	vR[i] = 1227;
}
else
{
	vR[i] = 1226;
}

}
else
{
	if(input[i]==(1225+offset1))
{
	vR[i] = 1225;
}
else
{
	vR[i] = 1224;
}

}

}

}
else
{
	if(input[i]>(1219+offset1))
{
	if(input[i]>(1221+offset1))
{
	if(input[i]==(1223+offset1))
{
	vR[i] = 1223;
}
else
{
	vR[i] = 1222;
}

}
else
{
	if(input[i]==(1221+offset1))
{
	vR[i] = 1221;
}
else
{
	vR[i] = 1220;
}

}

}
else
{
	if(input[i]>(1217+offset1))
{
	if(input[i]==(1219+offset1))
{
	vR[i] = 1219;
}
else
{
	vR[i] = 1218;
}

}
else
{
	if(input[i]==(1217+offset1))
{
	vR[i] = 1217;
}
else
{
	vR[i] = 1216;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1183+offset1))
{
	if(input[i]>(1199+offset1))
{
	if(input[i]>(1207+offset1))
{
	if(input[i]>(1211+offset1))
{
	if(input[i]>(1213+offset1))
{
	if(input[i]==(1215+offset1))
{
	vR[i] = 1215;
}
else
{
	vR[i] = 1214;
}

}
else
{
	if(input[i]==(1213+offset1))
{
	vR[i] = 1213;
}
else
{
	vR[i] = 1212;
}

}

}
else
{
	if(input[i]>(1209+offset1))
{
	if(input[i]==(1211+offset1))
{
	vR[i] = 1211;
}
else
{
	vR[i] = 1210;
}

}
else
{
	if(input[i]==(1209+offset1))
{
	vR[i] = 1209;
}
else
{
	vR[i] = 1208;
}

}

}

}
else
{
	if(input[i]>(1203+offset1))
{
	if(input[i]>(1205+offset1))
{
	if(input[i]==(1207+offset1))
{
	vR[i] = 1207;
}
else
{
	vR[i] = 1206;
}

}
else
{
	if(input[i]==(1205+offset1))
{
	vR[i] = 1205;
}
else
{
	vR[i] = 1204;
}

}

}
else
{
	if(input[i]>(1201+offset1))
{
	if(input[i]==(1203+offset1))
{
	vR[i] = 1203;
}
else
{
	vR[i] = 1202;
}

}
else
{
	if(input[i]==(1201+offset1))
{
	vR[i] = 1201;
}
else
{
	vR[i] = 1200;
}

}

}

}

}
else
{
	if(input[i]>(1191+offset1))
{
	if(input[i]>(1195+offset1))
{
	if(input[i]>(1197+offset1))
{
	if(input[i]==(1199+offset1))
{
	vR[i] = 1199;
}
else
{
	vR[i] = 1198;
}

}
else
{
	if(input[i]==(1197+offset1))
{
	vR[i] = 1197;
}
else
{
	vR[i] = 1196;
}

}

}
else
{
	if(input[i]>(1193+offset1))
{
	if(input[i]==(1195+offset1))
{
	vR[i] = 1195;
}
else
{
	vR[i] = 1194;
}

}
else
{
	if(input[i]==(1193+offset1))
{
	vR[i] = 1193;
}
else
{
	vR[i] = 1192;
}

}

}

}
else
{
	if(input[i]>(1187+offset1))
{
	if(input[i]>(1189+offset1))
{
	if(input[i]==(1191+offset1))
{
	vR[i] = 1191;
}
else
{
	vR[i] = 1190;
}

}
else
{
	if(input[i]==(1189+offset1))
{
	vR[i] = 1189;
}
else
{
	vR[i] = 1188;
}

}

}
else
{
	if(input[i]>(1185+offset1))
{
	if(input[i]==(1187+offset1))
{
	vR[i] = 1187;
}
else
{
	vR[i] = 1186;
}

}
else
{
	if(input[i]==(1185+offset1))
{
	vR[i] = 1185;
}
else
{
	vR[i] = 1184;
}

}

}

}

}

}
else
{
	if(input[i]>(1167+offset1))
{
	if(input[i]>(1175+offset1))
{
	if(input[i]>(1179+offset1))
{
	if(input[i]>(1181+offset1))
{
	if(input[i]==(1183+offset1))
{
	vR[i] = 1183;
}
else
{
	vR[i] = 1182;
}

}
else
{
	if(input[i]==(1181+offset1))
{
	vR[i] = 1181;
}
else
{
	vR[i] = 1180;
}

}

}
else
{
	if(input[i]>(1177+offset1))
{
	if(input[i]==(1179+offset1))
{
	vR[i] = 1179;
}
else
{
	vR[i] = 1178;
}

}
else
{
	if(input[i]==(1177+offset1))
{
	vR[i] = 1177;
}
else
{
	vR[i] = 1176;
}

}

}

}
else
{
	if(input[i]>(1171+offset1))
{
	if(input[i]>(1173+offset1))
{
	if(input[i]==(1175+offset1))
{
	vR[i] = 1175;
}
else
{
	vR[i] = 1174;
}

}
else
{
	if(input[i]==(1173+offset1))
{
	vR[i] = 1173;
}
else
{
	vR[i] = 1172;
}

}

}
else
{
	if(input[i]>(1169+offset1))
{
	if(input[i]==(1171+offset1))
{
	vR[i] = 1171;
}
else
{
	vR[i] = 1170;
}

}
else
{
	if(input[i]==(1169+offset1))
{
	vR[i] = 1169;
}
else
{
	vR[i] = 1168;
}

}

}

}

}
else
{
	if(input[i]>(1159+offset1))
{
	if(input[i]>(1163+offset1))
{
	if(input[i]>(1165+offset1))
{
	if(input[i]==(1167+offset1))
{
	vR[i] = 1167;
}
else
{
	vR[i] = 1166;
}

}
else
{
	if(input[i]==(1165+offset1))
{
	vR[i] = 1165;
}
else
{
	vR[i] = 1164;
}

}

}
else
{
	if(input[i]>(1161+offset1))
{
	if(input[i]==(1163+offset1))
{
	vR[i] = 1163;
}
else
{
	vR[i] = 1162;
}

}
else
{
	if(input[i]==(1161+offset1))
{
	vR[i] = 1161;
}
else
{
	vR[i] = 1160;
}

}

}

}
else
{
	if(input[i]>(1155+offset1))
{
	if(input[i]>(1157+offset1))
{
	if(input[i]==(1159+offset1))
{
	vR[i] = 1159;
}
else
{
	vR[i] = 1158;
}

}
else
{
	if(input[i]==(1157+offset1))
{
	vR[i] = 1157;
}
else
{
	vR[i] = 1156;
}

}

}
else
{
	if(input[i]>(1153+offset1))
{
	if(input[i]==(1155+offset1))
{
	vR[i] = 1155;
}
else
{
	vR[i] = 1154;
}

}
else
{
	if(input[i]==(1153+offset1))
{
	vR[i] = 1153;
}
else
{
	vR[i] = 1152;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(1087+offset1))
{
	if(input[i]>(1119+offset1))
{
	if(input[i]>(1135+offset1))
{
	if(input[i]>(1143+offset1))
{
	if(input[i]>(1147+offset1))
{
	if(input[i]>(1149+offset1))
{
	if(input[i]==(1151+offset1))
{
	vR[i] = 1151;
}
else
{
	vR[i] = 1150;
}

}
else
{
	if(input[i]==(1149+offset1))
{
	vR[i] = 1149;
}
else
{
	vR[i] = 1148;
}

}

}
else
{
	if(input[i]>(1145+offset1))
{
	if(input[i]==(1147+offset1))
{
	vR[i] = 1147;
}
else
{
	vR[i] = 1146;
}

}
else
{
	if(input[i]==(1145+offset1))
{
	vR[i] = 1145;
}
else
{
	vR[i] = 1144;
}

}

}

}
else
{
	if(input[i]>(1139+offset1))
{
	if(input[i]>(1141+offset1))
{
	if(input[i]==(1143+offset1))
{
	vR[i] = 1143;
}
else
{
	vR[i] = 1142;
}

}
else
{
	if(input[i]==(1141+offset1))
{
	vR[i] = 1141;
}
else
{
	vR[i] = 1140;
}

}

}
else
{
	if(input[i]>(1137+offset1))
{
	if(input[i]==(1139+offset1))
{
	vR[i] = 1139;
}
else
{
	vR[i] = 1138;
}

}
else
{
	if(input[i]==(1137+offset1))
{
	vR[i] = 1137;
}
else
{
	vR[i] = 1136;
}

}

}

}

}
else
{
	if(input[i]>(1127+offset1))
{
	if(input[i]>(1131+offset1))
{
	if(input[i]>(1133+offset1))
{
	if(input[i]==(1135+offset1))
{
	vR[i] = 1135;
}
else
{
	vR[i] = 1134;
}

}
else
{
	if(input[i]==(1133+offset1))
{
	vR[i] = 1133;
}
else
{
	vR[i] = 1132;
}

}

}
else
{
	if(input[i]>(1129+offset1))
{
	if(input[i]==(1131+offset1))
{
	vR[i] = 1131;
}
else
{
	vR[i] = 1130;
}

}
else
{
	if(input[i]==(1129+offset1))
{
	vR[i] = 1129;
}
else
{
	vR[i] = 1128;
}

}

}

}
else
{
	if(input[i]>(1123+offset1))
{
	if(input[i]>(1125+offset1))
{
	if(input[i]==(1127+offset1))
{
	vR[i] = 1127;
}
else
{
	vR[i] = 1126;
}

}
else
{
	if(input[i]==(1125+offset1))
{
	vR[i] = 1125;
}
else
{
	vR[i] = 1124;
}

}

}
else
{
	if(input[i]>(1121+offset1))
{
	if(input[i]==(1123+offset1))
{
	vR[i] = 1123;
}
else
{
	vR[i] = 1122;
}

}
else
{
	if(input[i]==(1121+offset1))
{
	vR[i] = 1121;
}
else
{
	vR[i] = 1120;
}

}

}

}

}

}
else
{
	if(input[i]>(1103+offset1))
{
	if(input[i]>(1111+offset1))
{
	if(input[i]>(1115+offset1))
{
	if(input[i]>(1117+offset1))
{
	if(input[i]==(1119+offset1))
{
	vR[i] = 1119;
}
else
{
	vR[i] = 1118;
}

}
else
{
	if(input[i]==(1117+offset1))
{
	vR[i] = 1117;
}
else
{
	vR[i] = 1116;
}

}

}
else
{
	if(input[i]>(1113+offset1))
{
	if(input[i]==(1115+offset1))
{
	vR[i] = 1115;
}
else
{
	vR[i] = 1114;
}

}
else
{
	if(input[i]==(1113+offset1))
{
	vR[i] = 1113;
}
else
{
	vR[i] = 1112;
}

}

}

}
else
{
	if(input[i]>(1107+offset1))
{
	if(input[i]>(1109+offset1))
{
	if(input[i]==(1111+offset1))
{
	vR[i] = 1111;
}
else
{
	vR[i] = 1110;
}

}
else
{
	if(input[i]==(1109+offset1))
{
	vR[i] = 1109;
}
else
{
	vR[i] = 1108;
}

}

}
else
{
	if(input[i]>(1105+offset1))
{
	if(input[i]==(1107+offset1))
{
	vR[i] = 1107;
}
else
{
	vR[i] = 1106;
}

}
else
{
	if(input[i]==(1105+offset1))
{
	vR[i] = 1105;
}
else
{
	vR[i] = 1104;
}

}

}

}

}
else
{
	if(input[i]>(1095+offset1))
{
	if(input[i]>(1099+offset1))
{
	if(input[i]>(1101+offset1))
{
	if(input[i]==(1103+offset1))
{
	vR[i] = 1103;
}
else
{
	vR[i] = 1102;
}

}
else
{
	if(input[i]==(1101+offset1))
{
	vR[i] = 1101;
}
else
{
	vR[i] = 1100;
}

}

}
else
{
	if(input[i]>(1097+offset1))
{
	if(input[i]==(1099+offset1))
{
	vR[i] = 1099;
}
else
{
	vR[i] = 1098;
}

}
else
{
	if(input[i]==(1097+offset1))
{
	vR[i] = 1097;
}
else
{
	vR[i] = 1096;
}

}

}

}
else
{
	if(input[i]>(1091+offset1))
{
	if(input[i]>(1093+offset1))
{
	if(input[i]==(1095+offset1))
{
	vR[i] = 1095;
}
else
{
	vR[i] = 1094;
}

}
else
{
	if(input[i]==(1093+offset1))
{
	vR[i] = 1093;
}
else
{
	vR[i] = 1092;
}

}

}
else
{
	if(input[i]>(1089+offset1))
{
	if(input[i]==(1091+offset1))
{
	vR[i] = 1091;
}
else
{
	vR[i] = 1090;
}

}
else
{
	if(input[i]==(1089+offset1))
{
	vR[i] = 1089;
}
else
{
	vR[i] = 1088;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1055+offset1))
{
	if(input[i]>(1071+offset1))
{
	if(input[i]>(1079+offset1))
{
	if(input[i]>(1083+offset1))
{
	if(input[i]>(1085+offset1))
{
	if(input[i]==(1087+offset1))
{
	vR[i] = 1087;
}
else
{
	vR[i] = 1086;
}

}
else
{
	if(input[i]==(1085+offset1))
{
	vR[i] = 1085;
}
else
{
	vR[i] = 1084;
}

}

}
else
{
	if(input[i]>(1081+offset1))
{
	if(input[i]==(1083+offset1))
{
	vR[i] = 1083;
}
else
{
	vR[i] = 1082;
}

}
else
{
	if(input[i]==(1081+offset1))
{
	vR[i] = 1081;
}
else
{
	vR[i] = 1080;
}

}

}

}
else
{
	if(input[i]>(1075+offset1))
{
	if(input[i]>(1077+offset1))
{
	if(input[i]==(1079+offset1))
{
	vR[i] = 1079;
}
else
{
	vR[i] = 1078;
}

}
else
{
	if(input[i]==(1077+offset1))
{
	vR[i] = 1077;
}
else
{
	vR[i] = 1076;
}

}

}
else
{
	if(input[i]>(1073+offset1))
{
	if(input[i]==(1075+offset1))
{
	vR[i] = 1075;
}
else
{
	vR[i] = 1074;
}

}
else
{
	if(input[i]==(1073+offset1))
{
	vR[i] = 1073;
}
else
{
	vR[i] = 1072;
}

}

}

}

}
else
{
	if(input[i]>(1063+offset1))
{
	if(input[i]>(1067+offset1))
{
	if(input[i]>(1069+offset1))
{
	if(input[i]==(1071+offset1))
{
	vR[i] = 1071;
}
else
{
	vR[i] = 1070;
}

}
else
{
	if(input[i]==(1069+offset1))
{
	vR[i] = 1069;
}
else
{
	vR[i] = 1068;
}

}

}
else
{
	if(input[i]>(1065+offset1))
{
	if(input[i]==(1067+offset1))
{
	vR[i] = 1067;
}
else
{
	vR[i] = 1066;
}

}
else
{
	if(input[i]==(1065+offset1))
{
	vR[i] = 1065;
}
else
{
	vR[i] = 1064;
}

}

}

}
else
{
	if(input[i]>(1059+offset1))
{
	if(input[i]>(1061+offset1))
{
	if(input[i]==(1063+offset1))
{
	vR[i] = 1063;
}
else
{
	vR[i] = 1062;
}

}
else
{
	if(input[i]==(1061+offset1))
{
	vR[i] = 1061;
}
else
{
	vR[i] = 1060;
}

}

}
else
{
	if(input[i]>(1057+offset1))
{
	if(input[i]==(1059+offset1))
{
	vR[i] = 1059;
}
else
{
	vR[i] = 1058;
}

}
else
{
	if(input[i]==(1057+offset1))
{
	vR[i] = 1057;
}
else
{
	vR[i] = 1056;
}

}

}

}

}

}
else
{
	if(input[i]>(1039+offset1))
{
	if(input[i]>(1047+offset1))
{
	if(input[i]>(1051+offset1))
{
	if(input[i]>(1053+offset1))
{
	if(input[i]==(1055+offset1))
{
	vR[i] = 1055;
}
else
{
	vR[i] = 1054;
}

}
else
{
	if(input[i]==(1053+offset1))
{
	vR[i] = 1053;
}
else
{
	vR[i] = 1052;
}

}

}
else
{
	if(input[i]>(1049+offset1))
{
	if(input[i]==(1051+offset1))
{
	vR[i] = 1051;
}
else
{
	vR[i] = 1050;
}

}
else
{
	if(input[i]==(1049+offset1))
{
	vR[i] = 1049;
}
else
{
	vR[i] = 1048;
}

}

}

}
else
{
	if(input[i]>(1043+offset1))
{
	if(input[i]>(1045+offset1))
{
	if(input[i]==(1047+offset1))
{
	vR[i] = 1047;
}
else
{
	vR[i] = 1046;
}

}
else
{
	if(input[i]==(1045+offset1))
{
	vR[i] = 1045;
}
else
{
	vR[i] = 1044;
}

}

}
else
{
	if(input[i]>(1041+offset1))
{
	if(input[i]==(1043+offset1))
{
	vR[i] = 1043;
}
else
{
	vR[i] = 1042;
}

}
else
{
	if(input[i]==(1041+offset1))
{
	vR[i] = 1041;
}
else
{
	vR[i] = 1040;
}

}

}

}

}
else
{
	if(input[i]>(1031+offset1))
{
	if(input[i]>(1035+offset1))
{
	if(input[i]>(1037+offset1))
{
	if(input[i]==(1039+offset1))
{
	vR[i] = 1039;
}
else
{
	vR[i] = 1038;
}

}
else
{
	if(input[i]==(1037+offset1))
{
	vR[i] = 1037;
}
else
{
	vR[i] = 1036;
}

}

}
else
{
	if(input[i]>(1033+offset1))
{
	if(input[i]==(1035+offset1))
{
	vR[i] = 1035;
}
else
{
	vR[i] = 1034;
}

}
else
{
	if(input[i]==(1033+offset1))
{
	vR[i] = 1033;
}
else
{
	vR[i] = 1032;
}

}

}

}
else
{
	if(input[i]>(1027+offset1))
{
	if(input[i]>(1029+offset1))
{
	if(input[i]==(1031+offset1))
{
	vR[i] = 1031;
}
else
{
	vR[i] = 1030;
}

}
else
{
	if(input[i]==(1029+offset1))
{
	vR[i] = 1029;
}
else
{
	vR[i] = 1028;
}

}

}
else
{
	if(input[i]>(1025+offset1))
{
	if(input[i]==(1027+offset1))
{
	vR[i] = 1027;
}
else
{
	vR[i] = 1026;
}

}
else
{
	if(input[i]==(1025+offset1))
{
	vR[i] = 1025;
}
else
{
	vR[i] = 1024;
}

}

}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(511+offset1))
{
	if(input[i]>(767+offset1))
{
	if(input[i]>(895+offset1))
{
	if(input[i]>(959+offset1))
{
	if(input[i]>(991+offset1))
{
	if(input[i]>(1007+offset1))
{
	if(input[i]>(1015+offset1))
{
	if(input[i]>(1019+offset1))
{
	if(input[i]>(1021+offset1))
{
	if(input[i]==(1023+offset1))
{
	vR[i] = 1023;
}
else
{
	vR[i] = 1022;
}

}
else
{
	if(input[i]==(1021+offset1))
{
	vR[i] = 1021;
}
else
{
	vR[i] = 1020;
}

}

}
else
{
	if(input[i]>(1017+offset1))
{
	if(input[i]==(1019+offset1))
{
	vR[i] = 1019;
}
else
{
	vR[i] = 1018;
}

}
else
{
	if(input[i]==(1017+offset1))
{
	vR[i] = 1017;
}
else
{
	vR[i] = 1016;
}

}

}

}
else
{
	if(input[i]>(1011+offset1))
{
	if(input[i]>(1013+offset1))
{
	if(input[i]==(1015+offset1))
{
	vR[i] = 1015;
}
else
{
	vR[i] = 1014;
}

}
else
{
	if(input[i]==(1013+offset1))
{
	vR[i] = 1013;
}
else
{
	vR[i] = 1012;
}

}

}
else
{
	if(input[i]>(1009+offset1))
{
	if(input[i]==(1011+offset1))
{
	vR[i] = 1011;
}
else
{
	vR[i] = 1010;
}

}
else
{
	if(input[i]==(1009+offset1))
{
	vR[i] = 1009;
}
else
{
	vR[i] = 1008;
}

}

}

}

}
else
{
	if(input[i]>(999+offset1))
{
	if(input[i]>(1003+offset1))
{
	if(input[i]>(1005+offset1))
{
	if(input[i]==(1007+offset1))
{
	vR[i] = 1007;
}
else
{
	vR[i] = 1006;
}

}
else
{
	if(input[i]==(1005+offset1))
{
	vR[i] = 1005;
}
else
{
	vR[i] = 1004;
}

}

}
else
{
	if(input[i]>(1001+offset1))
{
	if(input[i]==(1003+offset1))
{
	vR[i] = 1003;
}
else
{
	vR[i] = 1002;
}

}
else
{
	if(input[i]==(1001+offset1))
{
	vR[i] = 1001;
}
else
{
	vR[i] = 1000;
}

}

}

}
else
{
	if(input[i]>(995+offset1))
{
	if(input[i]>(997+offset1))
{
	if(input[i]==(999+offset1))
{
	vR[i] = 999;
}
else
{
	vR[i] = 998;
}

}
else
{
	if(input[i]==(997+offset1))
{
	vR[i] = 997;
}
else
{
	vR[i] = 996;
}

}

}
else
{
	if(input[i]>(993+offset1))
{
	if(input[i]==(995+offset1))
{
	vR[i] = 995;
}
else
{
	vR[i] = 994;
}

}
else
{
	if(input[i]==(993+offset1))
{
	vR[i] = 993;
}
else
{
	vR[i] = 992;
}

}

}

}

}

}
else
{
	if(input[i]>(975+offset1))
{
	if(input[i]>(983+offset1))
{
	if(input[i]>(987+offset1))
{
	if(input[i]>(989+offset1))
{
	if(input[i]==(991+offset1))
{
	vR[i] = 991;
}
else
{
	vR[i] = 990;
}

}
else
{
	if(input[i]==(989+offset1))
{
	vR[i] = 989;
}
else
{
	vR[i] = 988;
}

}

}
else
{
	if(input[i]>(985+offset1))
{
	if(input[i]==(987+offset1))
{
	vR[i] = 987;
}
else
{
	vR[i] = 986;
}

}
else
{
	if(input[i]==(985+offset1))
{
	vR[i] = 985;
}
else
{
	vR[i] = 984;
}

}

}

}
else
{
	if(input[i]>(979+offset1))
{
	if(input[i]>(981+offset1))
{
	if(input[i]==(983+offset1))
{
	vR[i] = 983;
}
else
{
	vR[i] = 982;
}

}
else
{
	if(input[i]==(981+offset1))
{
	vR[i] = 981;
}
else
{
	vR[i] = 980;
}

}

}
else
{
	if(input[i]>(977+offset1))
{
	if(input[i]==(979+offset1))
{
	vR[i] = 979;
}
else
{
	vR[i] = 978;
}

}
else
{
	if(input[i]==(977+offset1))
{
	vR[i] = 977;
}
else
{
	vR[i] = 976;
}

}

}

}

}
else
{
	if(input[i]>(967+offset1))
{
	if(input[i]>(971+offset1))
{
	if(input[i]>(973+offset1))
{
	if(input[i]==(975+offset1))
{
	vR[i] = 975;
}
else
{
	vR[i] = 974;
}

}
else
{
	if(input[i]==(973+offset1))
{
	vR[i] = 973;
}
else
{
	vR[i] = 972;
}

}

}
else
{
	if(input[i]>(969+offset1))
{
	if(input[i]==(971+offset1))
{
	vR[i] = 971;
}
else
{
	vR[i] = 970;
}

}
else
{
	if(input[i]==(969+offset1))
{
	vR[i] = 969;
}
else
{
	vR[i] = 968;
}

}

}

}
else
{
	if(input[i]>(963+offset1))
{
	if(input[i]>(965+offset1))
{
	if(input[i]==(967+offset1))
{
	vR[i] = 967;
}
else
{
	vR[i] = 966;
}

}
else
{
	if(input[i]==(965+offset1))
{
	vR[i] = 965;
}
else
{
	vR[i] = 964;
}

}

}
else
{
	if(input[i]>(961+offset1))
{
	if(input[i]==(963+offset1))
{
	vR[i] = 963;
}
else
{
	vR[i] = 962;
}

}
else
{
	if(input[i]==(961+offset1))
{
	vR[i] = 961;
}
else
{
	vR[i] = 960;
}

}

}

}

}

}

}
else
{
	if(input[i]>(927+offset1))
{
	if(input[i]>(943+offset1))
{
	if(input[i]>(951+offset1))
{
	if(input[i]>(955+offset1))
{
	if(input[i]>(957+offset1))
{
	if(input[i]==(959+offset1))
{
	vR[i] = 959;
}
else
{
	vR[i] = 958;
}

}
else
{
	if(input[i]==(957+offset1))
{
	vR[i] = 957;
}
else
{
	vR[i] = 956;
}

}

}
else
{
	if(input[i]>(953+offset1))
{
	if(input[i]==(955+offset1))
{
	vR[i] = 955;
}
else
{
	vR[i] = 954;
}

}
else
{
	if(input[i]==(953+offset1))
{
	vR[i] = 953;
}
else
{
	vR[i] = 952;
}

}

}

}
else
{
	if(input[i]>(947+offset1))
{
	if(input[i]>(949+offset1))
{
	if(input[i]==(951+offset1))
{
	vR[i] = 951;
}
else
{
	vR[i] = 950;
}

}
else
{
	if(input[i]==(949+offset1))
{
	vR[i] = 949;
}
else
{
	vR[i] = 948;
}

}

}
else
{
	if(input[i]>(945+offset1))
{
	if(input[i]==(947+offset1))
{
	vR[i] = 947;
}
else
{
	vR[i] = 946;
}

}
else
{
	if(input[i]==(945+offset1))
{
	vR[i] = 945;
}
else
{
	vR[i] = 944;
}

}

}

}

}
else
{
	if(input[i]>(935+offset1))
{
	if(input[i]>(939+offset1))
{
	if(input[i]>(941+offset1))
{
	if(input[i]==(943+offset1))
{
	vR[i] = 943;
}
else
{
	vR[i] = 942;
}

}
else
{
	if(input[i]==(941+offset1))
{
	vR[i] = 941;
}
else
{
	vR[i] = 940;
}

}

}
else
{
	if(input[i]>(937+offset1))
{
	if(input[i]==(939+offset1))
{
	vR[i] = 939;
}
else
{
	vR[i] = 938;
}

}
else
{
	if(input[i]==(937+offset1))
{
	vR[i] = 937;
}
else
{
	vR[i] = 936;
}

}

}

}
else
{
	if(input[i]>(931+offset1))
{
	if(input[i]>(933+offset1))
{
	if(input[i]==(935+offset1))
{
	vR[i] = 935;
}
else
{
	vR[i] = 934;
}

}
else
{
	if(input[i]==(933+offset1))
{
	vR[i] = 933;
}
else
{
	vR[i] = 932;
}

}

}
else
{
	if(input[i]>(929+offset1))
{
	if(input[i]==(931+offset1))
{
	vR[i] = 931;
}
else
{
	vR[i] = 930;
}

}
else
{
	if(input[i]==(929+offset1))
{
	vR[i] = 929;
}
else
{
	vR[i] = 928;
}

}

}

}

}

}
else
{
	if(input[i]>(911+offset1))
{
	if(input[i]>(919+offset1))
{
	if(input[i]>(923+offset1))
{
	if(input[i]>(925+offset1))
{
	if(input[i]==(927+offset1))
{
	vR[i] = 927;
}
else
{
	vR[i] = 926;
}

}
else
{
	if(input[i]==(925+offset1))
{
	vR[i] = 925;
}
else
{
	vR[i] = 924;
}

}

}
else
{
	if(input[i]>(921+offset1))
{
	if(input[i]==(923+offset1))
{
	vR[i] = 923;
}
else
{
	vR[i] = 922;
}

}
else
{
	if(input[i]==(921+offset1))
{
	vR[i] = 921;
}
else
{
	vR[i] = 920;
}

}

}

}
else
{
	if(input[i]>(915+offset1))
{
	if(input[i]>(917+offset1))
{
	if(input[i]==(919+offset1))
{
	vR[i] = 919;
}
else
{
	vR[i] = 918;
}

}
else
{
	if(input[i]==(917+offset1))
{
	vR[i] = 917;
}
else
{
	vR[i] = 916;
}

}

}
else
{
	if(input[i]>(913+offset1))
{
	if(input[i]==(915+offset1))
{
	vR[i] = 915;
}
else
{
	vR[i] = 914;
}

}
else
{
	if(input[i]==(913+offset1))
{
	vR[i] = 913;
}
else
{
	vR[i] = 912;
}

}

}

}

}
else
{
	if(input[i]>(903+offset1))
{
	if(input[i]>(907+offset1))
{
	if(input[i]>(909+offset1))
{
	if(input[i]==(911+offset1))
{
	vR[i] = 911;
}
else
{
	vR[i] = 910;
}

}
else
{
	if(input[i]==(909+offset1))
{
	vR[i] = 909;
}
else
{
	vR[i] = 908;
}

}

}
else
{
	if(input[i]>(905+offset1))
{
	if(input[i]==(907+offset1))
{
	vR[i] = 907;
}
else
{
	vR[i] = 906;
}

}
else
{
	if(input[i]==(905+offset1))
{
	vR[i] = 905;
}
else
{
	vR[i] = 904;
}

}

}

}
else
{
	if(input[i]>(899+offset1))
{
	if(input[i]>(901+offset1))
{
	if(input[i]==(903+offset1))
{
	vR[i] = 903;
}
else
{
	vR[i] = 902;
}

}
else
{
	if(input[i]==(901+offset1))
{
	vR[i] = 901;
}
else
{
	vR[i] = 900;
}

}

}
else
{
	if(input[i]>(897+offset1))
{
	if(input[i]==(899+offset1))
{
	vR[i] = 899;
}
else
{
	vR[i] = 898;
}

}
else
{
	if(input[i]==(897+offset1))
{
	vR[i] = 897;
}
else
{
	vR[i] = 896;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(831+offset1))
{
	if(input[i]>(863+offset1))
{
	if(input[i]>(879+offset1))
{
	if(input[i]>(887+offset1))
{
	if(input[i]>(891+offset1))
{
	if(input[i]>(893+offset1))
{
	if(input[i]==(895+offset1))
{
	vR[i] = 895;
}
else
{
	vR[i] = 894;
}

}
else
{
	if(input[i]==(893+offset1))
{
	vR[i] = 893;
}
else
{
	vR[i] = 892;
}

}

}
else
{
	if(input[i]>(889+offset1))
{
	if(input[i]==(891+offset1))
{
	vR[i] = 891;
}
else
{
	vR[i] = 890;
}

}
else
{
	if(input[i]==(889+offset1))
{
	vR[i] = 889;
}
else
{
	vR[i] = 888;
}

}

}

}
else
{
	if(input[i]>(883+offset1))
{
	if(input[i]>(885+offset1))
{
	if(input[i]==(887+offset1))
{
	vR[i] = 887;
}
else
{
	vR[i] = 886;
}

}
else
{
	if(input[i]==(885+offset1))
{
	vR[i] = 885;
}
else
{
	vR[i] = 884;
}

}

}
else
{
	if(input[i]>(881+offset1))
{
	if(input[i]==(883+offset1))
{
	vR[i] = 883;
}
else
{
	vR[i] = 882;
}

}
else
{
	if(input[i]==(881+offset1))
{
	vR[i] = 881;
}
else
{
	vR[i] = 880;
}

}

}

}

}
else
{
	if(input[i]>(871+offset1))
{
	if(input[i]>(875+offset1))
{
	if(input[i]>(877+offset1))
{
	if(input[i]==(879+offset1))
{
	vR[i] = 879;
}
else
{
	vR[i] = 878;
}

}
else
{
	if(input[i]==(877+offset1))
{
	vR[i] = 877;
}
else
{
	vR[i] = 876;
}

}

}
else
{
	if(input[i]>(873+offset1))
{
	if(input[i]==(875+offset1))
{
	vR[i] = 875;
}
else
{
	vR[i] = 874;
}

}
else
{
	if(input[i]==(873+offset1))
{
	vR[i] = 873;
}
else
{
	vR[i] = 872;
}

}

}

}
else
{
	if(input[i]>(867+offset1))
{
	if(input[i]>(869+offset1))
{
	if(input[i]==(871+offset1))
{
	vR[i] = 871;
}
else
{
	vR[i] = 870;
}

}
else
{
	if(input[i]==(869+offset1))
{
	vR[i] = 869;
}
else
{
	vR[i] = 868;
}

}

}
else
{
	if(input[i]>(865+offset1))
{
	if(input[i]==(867+offset1))
{
	vR[i] = 867;
}
else
{
	vR[i] = 866;
}

}
else
{
	if(input[i]==(865+offset1))
{
	vR[i] = 865;
}
else
{
	vR[i] = 864;
}

}

}

}

}

}
else
{
	if(input[i]>(847+offset1))
{
	if(input[i]>(855+offset1))
{
	if(input[i]>(859+offset1))
{
	if(input[i]>(861+offset1))
{
	if(input[i]==(863+offset1))
{
	vR[i] = 863;
}
else
{
	vR[i] = 862;
}

}
else
{
	if(input[i]==(861+offset1))
{
	vR[i] = 861;
}
else
{
	vR[i] = 860;
}

}

}
else
{
	if(input[i]>(857+offset1))
{
	if(input[i]==(859+offset1))
{
	vR[i] = 859;
}
else
{
	vR[i] = 858;
}

}
else
{
	if(input[i]==(857+offset1))
{
	vR[i] = 857;
}
else
{
	vR[i] = 856;
}

}

}

}
else
{
	if(input[i]>(851+offset1))
{
	if(input[i]>(853+offset1))
{
	if(input[i]==(855+offset1))
{
	vR[i] = 855;
}
else
{
	vR[i] = 854;
}

}
else
{
	if(input[i]==(853+offset1))
{
	vR[i] = 853;
}
else
{
	vR[i] = 852;
}

}

}
else
{
	if(input[i]>(849+offset1))
{
	if(input[i]==(851+offset1))
{
	vR[i] = 851;
}
else
{
	vR[i] = 850;
}

}
else
{
	if(input[i]==(849+offset1))
{
	vR[i] = 849;
}
else
{
	vR[i] = 848;
}

}

}

}

}
else
{
	if(input[i]>(839+offset1))
{
	if(input[i]>(843+offset1))
{
	if(input[i]>(845+offset1))
{
	if(input[i]==(847+offset1))
{
	vR[i] = 847;
}
else
{
	vR[i] = 846;
}

}
else
{
	if(input[i]==(845+offset1))
{
	vR[i] = 845;
}
else
{
	vR[i] = 844;
}

}

}
else
{
	if(input[i]>(841+offset1))
{
	if(input[i]==(843+offset1))
{
	vR[i] = 843;
}
else
{
	vR[i] = 842;
}

}
else
{
	if(input[i]==(841+offset1))
{
	vR[i] = 841;
}
else
{
	vR[i] = 840;
}

}

}

}
else
{
	if(input[i]>(835+offset1))
{
	if(input[i]>(837+offset1))
{
	if(input[i]==(839+offset1))
{
	vR[i] = 839;
}
else
{
	vR[i] = 838;
}

}
else
{
	if(input[i]==(837+offset1))
{
	vR[i] = 837;
}
else
{
	vR[i] = 836;
}

}

}
else
{
	if(input[i]>(833+offset1))
{
	if(input[i]==(835+offset1))
{
	vR[i] = 835;
}
else
{
	vR[i] = 834;
}

}
else
{
	if(input[i]==(833+offset1))
{
	vR[i] = 833;
}
else
{
	vR[i] = 832;
}

}

}

}

}

}

}
else
{
	if(input[i]>(799+offset1))
{
	if(input[i]>(815+offset1))
{
	if(input[i]>(823+offset1))
{
	if(input[i]>(827+offset1))
{
	if(input[i]>(829+offset1))
{
	if(input[i]==(831+offset1))
{
	vR[i] = 831;
}
else
{
	vR[i] = 830;
}

}
else
{
	if(input[i]==(829+offset1))
{
	vR[i] = 829;
}
else
{
	vR[i] = 828;
}

}

}
else
{
	if(input[i]>(825+offset1))
{
	if(input[i]==(827+offset1))
{
	vR[i] = 827;
}
else
{
	vR[i] = 826;
}

}
else
{
	if(input[i]==(825+offset1))
{
	vR[i] = 825;
}
else
{
	vR[i] = 824;
}

}

}

}
else
{
	if(input[i]>(819+offset1))
{
	if(input[i]>(821+offset1))
{
	if(input[i]==(823+offset1))
{
	vR[i] = 823;
}
else
{
	vR[i] = 822;
}

}
else
{
	if(input[i]==(821+offset1))
{
	vR[i] = 821;
}
else
{
	vR[i] = 820;
}

}

}
else
{
	if(input[i]>(817+offset1))
{
	if(input[i]==(819+offset1))
{
	vR[i] = 819;
}
else
{
	vR[i] = 818;
}

}
else
{
	if(input[i]==(817+offset1))
{
	vR[i] = 817;
}
else
{
	vR[i] = 816;
}

}

}

}

}
else
{
	if(input[i]>(807+offset1))
{
	if(input[i]>(811+offset1))
{
	if(input[i]>(813+offset1))
{
	if(input[i]==(815+offset1))
{
	vR[i] = 815;
}
else
{
	vR[i] = 814;
}

}
else
{
	if(input[i]==(813+offset1))
{
	vR[i] = 813;
}
else
{
	vR[i] = 812;
}

}

}
else
{
	if(input[i]>(809+offset1))
{
	if(input[i]==(811+offset1))
{
	vR[i] = 811;
}
else
{
	vR[i] = 810;
}

}
else
{
	if(input[i]==(809+offset1))
{
	vR[i] = 809;
}
else
{
	vR[i] = 808;
}

}

}

}
else
{
	if(input[i]>(803+offset1))
{
	if(input[i]>(805+offset1))
{
	if(input[i]==(807+offset1))
{
	vR[i] = 807;
}
else
{
	vR[i] = 806;
}

}
else
{
	if(input[i]==(805+offset1))
{
	vR[i] = 805;
}
else
{
	vR[i] = 804;
}

}

}
else
{
	if(input[i]>(801+offset1))
{
	if(input[i]==(803+offset1))
{
	vR[i] = 803;
}
else
{
	vR[i] = 802;
}

}
else
{
	if(input[i]==(801+offset1))
{
	vR[i] = 801;
}
else
{
	vR[i] = 800;
}

}

}

}

}

}
else
{
	if(input[i]>(783+offset1))
{
	if(input[i]>(791+offset1))
{
	if(input[i]>(795+offset1))
{
	if(input[i]>(797+offset1))
{
	if(input[i]==(799+offset1))
{
	vR[i] = 799;
}
else
{
	vR[i] = 798;
}

}
else
{
	if(input[i]==(797+offset1))
{
	vR[i] = 797;
}
else
{
	vR[i] = 796;
}

}

}
else
{
	if(input[i]>(793+offset1))
{
	if(input[i]==(795+offset1))
{
	vR[i] = 795;
}
else
{
	vR[i] = 794;
}

}
else
{
	if(input[i]==(793+offset1))
{
	vR[i] = 793;
}
else
{
	vR[i] = 792;
}

}

}

}
else
{
	if(input[i]>(787+offset1))
{
	if(input[i]>(789+offset1))
{
	if(input[i]==(791+offset1))
{
	vR[i] = 791;
}
else
{
	vR[i] = 790;
}

}
else
{
	if(input[i]==(789+offset1))
{
	vR[i] = 789;
}
else
{
	vR[i] = 788;
}

}

}
else
{
	if(input[i]>(785+offset1))
{
	if(input[i]==(787+offset1))
{
	vR[i] = 787;
}
else
{
	vR[i] = 786;
}

}
else
{
	if(input[i]==(785+offset1))
{
	vR[i] = 785;
}
else
{
	vR[i] = 784;
}

}

}

}

}
else
{
	if(input[i]>(775+offset1))
{
	if(input[i]>(779+offset1))
{
	if(input[i]>(781+offset1))
{
	if(input[i]==(783+offset1))
{
	vR[i] = 783;
}
else
{
	vR[i] = 782;
}

}
else
{
	if(input[i]==(781+offset1))
{
	vR[i] = 781;
}
else
{
	vR[i] = 780;
}

}

}
else
{
	if(input[i]>(777+offset1))
{
	if(input[i]==(779+offset1))
{
	vR[i] = 779;
}
else
{
	vR[i] = 778;
}

}
else
{
	if(input[i]==(777+offset1))
{
	vR[i] = 777;
}
else
{
	vR[i] = 776;
}

}

}

}
else
{
	if(input[i]>(771+offset1))
{
	if(input[i]>(773+offset1))
{
	if(input[i]==(775+offset1))
{
	vR[i] = 775;
}
else
{
	vR[i] = 774;
}

}
else
{
	if(input[i]==(773+offset1))
{
	vR[i] = 773;
}
else
{
	vR[i] = 772;
}

}

}
else
{
	if(input[i]>(769+offset1))
{
	if(input[i]==(771+offset1))
{
	vR[i] = 771;
}
else
{
	vR[i] = 770;
}

}
else
{
	if(input[i]==(769+offset1))
{
	vR[i] = 769;
}
else
{
	vR[i] = 768;
}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(639+offset1))
{
	if(input[i]>(703+offset1))
{
	if(input[i]>(735+offset1))
{
	if(input[i]>(751+offset1))
{
	if(input[i]>(759+offset1))
{
	if(input[i]>(763+offset1))
{
	if(input[i]>(765+offset1))
{
	if(input[i]==(767+offset1))
{
	vR[i] = 767;
}
else
{
	vR[i] = 766;
}

}
else
{
	if(input[i]==(765+offset1))
{
	vR[i] = 765;
}
else
{
	vR[i] = 764;
}

}

}
else
{
	if(input[i]>(761+offset1))
{
	if(input[i]==(763+offset1))
{
	vR[i] = 763;
}
else
{
	vR[i] = 762;
}

}
else
{
	if(input[i]==(761+offset1))
{
	vR[i] = 761;
}
else
{
	vR[i] = 760;
}

}

}

}
else
{
	if(input[i]>(755+offset1))
{
	if(input[i]>(757+offset1))
{
	if(input[i]==(759+offset1))
{
	vR[i] = 759;
}
else
{
	vR[i] = 758;
}

}
else
{
	if(input[i]==(757+offset1))
{
	vR[i] = 757;
}
else
{
	vR[i] = 756;
}

}

}
else
{
	if(input[i]>(753+offset1))
{
	if(input[i]==(755+offset1))
{
	vR[i] = 755;
}
else
{
	vR[i] = 754;
}

}
else
{
	if(input[i]==(753+offset1))
{
	vR[i] = 753;
}
else
{
	vR[i] = 752;
}

}

}

}

}
else
{
	if(input[i]>(743+offset1))
{
	if(input[i]>(747+offset1))
{
	if(input[i]>(749+offset1))
{
	if(input[i]==(751+offset1))
{
	vR[i] = 751;
}
else
{
	vR[i] = 750;
}

}
else
{
	if(input[i]==(749+offset1))
{
	vR[i] = 749;
}
else
{
	vR[i] = 748;
}

}

}
else
{
	if(input[i]>(745+offset1))
{
	if(input[i]==(747+offset1))
{
	vR[i] = 747;
}
else
{
	vR[i] = 746;
}

}
else
{
	if(input[i]==(745+offset1))
{
	vR[i] = 745;
}
else
{
	vR[i] = 744;
}

}

}

}
else
{
	if(input[i]>(739+offset1))
{
	if(input[i]>(741+offset1))
{
	if(input[i]==(743+offset1))
{
	vR[i] = 743;
}
else
{
	vR[i] = 742;
}

}
else
{
	if(input[i]==(741+offset1))
{
	vR[i] = 741;
}
else
{
	vR[i] = 740;
}

}

}
else
{
	if(input[i]>(737+offset1))
{
	if(input[i]==(739+offset1))
{
	vR[i] = 739;
}
else
{
	vR[i] = 738;
}

}
else
{
	if(input[i]==(737+offset1))
{
	vR[i] = 737;
}
else
{
	vR[i] = 736;
}

}

}

}

}

}
else
{
	if(input[i]>(719+offset1))
{
	if(input[i]>(727+offset1))
{
	if(input[i]>(731+offset1))
{
	if(input[i]>(733+offset1))
{
	if(input[i]==(735+offset1))
{
	vR[i] = 735;
}
else
{
	vR[i] = 734;
}

}
else
{
	if(input[i]==(733+offset1))
{
	vR[i] = 733;
}
else
{
	vR[i] = 732;
}

}

}
else
{
	if(input[i]>(729+offset1))
{
	if(input[i]==(731+offset1))
{
	vR[i] = 731;
}
else
{
	vR[i] = 730;
}

}
else
{
	if(input[i]==(729+offset1))
{
	vR[i] = 729;
}
else
{
	vR[i] = 728;
}

}

}

}
else
{
	if(input[i]>(723+offset1))
{
	if(input[i]>(725+offset1))
{
	if(input[i]==(727+offset1))
{
	vR[i] = 727;
}
else
{
	vR[i] = 726;
}

}
else
{
	if(input[i]==(725+offset1))
{
	vR[i] = 725;
}
else
{
	vR[i] = 724;
}

}

}
else
{
	if(input[i]>(721+offset1))
{
	if(input[i]==(723+offset1))
{
	vR[i] = 723;
}
else
{
	vR[i] = 722;
}

}
else
{
	if(input[i]==(721+offset1))
{
	vR[i] = 721;
}
else
{
	vR[i] = 720;
}

}

}

}

}
else
{
	if(input[i]>(711+offset1))
{
	if(input[i]>(715+offset1))
{
	if(input[i]>(717+offset1))
{
	if(input[i]==(719+offset1))
{
	vR[i] = 719;
}
else
{
	vR[i] = 718;
}

}
else
{
	if(input[i]==(717+offset1))
{
	vR[i] = 717;
}
else
{
	vR[i] = 716;
}

}

}
else
{
	if(input[i]>(713+offset1))
{
	if(input[i]==(715+offset1))
{
	vR[i] = 715;
}
else
{
	vR[i] = 714;
}

}
else
{
	if(input[i]==(713+offset1))
{
	vR[i] = 713;
}
else
{
	vR[i] = 712;
}

}

}

}
else
{
	if(input[i]>(707+offset1))
{
	if(input[i]>(709+offset1))
{
	if(input[i]==(711+offset1))
{
	vR[i] = 711;
}
else
{
	vR[i] = 710;
}

}
else
{
	if(input[i]==(709+offset1))
{
	vR[i] = 709;
}
else
{
	vR[i] = 708;
}

}

}
else
{
	if(input[i]>(705+offset1))
{
	if(input[i]==(707+offset1))
{
	vR[i] = 707;
}
else
{
	vR[i] = 706;
}

}
else
{
	if(input[i]==(705+offset1))
{
	vR[i] = 705;
}
else
{
	vR[i] = 704;
}

}

}

}

}

}

}
else
{
	if(input[i]>(671+offset1))
{
	if(input[i]>(687+offset1))
{
	if(input[i]>(695+offset1))
{
	if(input[i]>(699+offset1))
{
	if(input[i]>(701+offset1))
{
	if(input[i]==(703+offset1))
{
	vR[i] = 703;
}
else
{
	vR[i] = 702;
}

}
else
{
	if(input[i]==(701+offset1))
{
	vR[i] = 701;
}
else
{
	vR[i] = 700;
}

}

}
else
{
	if(input[i]>(697+offset1))
{
	if(input[i]==(699+offset1))
{
	vR[i] = 699;
}
else
{
	vR[i] = 698;
}

}
else
{
	if(input[i]==(697+offset1))
{
	vR[i] = 697;
}
else
{
	vR[i] = 696;
}

}

}

}
else
{
	if(input[i]>(691+offset1))
{
	if(input[i]>(693+offset1))
{
	if(input[i]==(695+offset1))
{
	vR[i] = 695;
}
else
{
	vR[i] = 694;
}

}
else
{
	if(input[i]==(693+offset1))
{
	vR[i] = 693;
}
else
{
	vR[i] = 692;
}

}

}
else
{
	if(input[i]>(689+offset1))
{
	if(input[i]==(691+offset1))
{
	vR[i] = 691;
}
else
{
	vR[i] = 690;
}

}
else
{
	if(input[i]==(689+offset1))
{
	vR[i] = 689;
}
else
{
	vR[i] = 688;
}

}

}

}

}
else
{
	if(input[i]>(679+offset1))
{
	if(input[i]>(683+offset1))
{
	if(input[i]>(685+offset1))
{
	if(input[i]==(687+offset1))
{
	vR[i] = 687;
}
else
{
	vR[i] = 686;
}

}
else
{
	if(input[i]==(685+offset1))
{
	vR[i] = 685;
}
else
{
	vR[i] = 684;
}

}

}
else
{
	if(input[i]>(681+offset1))
{
	if(input[i]==(683+offset1))
{
	vR[i] = 683;
}
else
{
	vR[i] = 682;
}

}
else
{
	if(input[i]==(681+offset1))
{
	vR[i] = 681;
}
else
{
	vR[i] = 680;
}

}

}

}
else
{
	if(input[i]>(675+offset1))
{
	if(input[i]>(677+offset1))
{
	if(input[i]==(679+offset1))
{
	vR[i] = 679;
}
else
{
	vR[i] = 678;
}

}
else
{
	if(input[i]==(677+offset1))
{
	vR[i] = 677;
}
else
{
	vR[i] = 676;
}

}

}
else
{
	if(input[i]>(673+offset1))
{
	if(input[i]==(675+offset1))
{
	vR[i] = 675;
}
else
{
	vR[i] = 674;
}

}
else
{
	if(input[i]==(673+offset1))
{
	vR[i] = 673;
}
else
{
	vR[i] = 672;
}

}

}

}

}

}
else
{
	if(input[i]>(655+offset1))
{
	if(input[i]>(663+offset1))
{
	if(input[i]>(667+offset1))
{
	if(input[i]>(669+offset1))
{
	if(input[i]==(671+offset1))
{
	vR[i] = 671;
}
else
{
	vR[i] = 670;
}

}
else
{
	if(input[i]==(669+offset1))
{
	vR[i] = 669;
}
else
{
	vR[i] = 668;
}

}

}
else
{
	if(input[i]>(665+offset1))
{
	if(input[i]==(667+offset1))
{
	vR[i] = 667;
}
else
{
	vR[i] = 666;
}

}
else
{
	if(input[i]==(665+offset1))
{
	vR[i] = 665;
}
else
{
	vR[i] = 664;
}

}

}

}
else
{
	if(input[i]>(659+offset1))
{
	if(input[i]>(661+offset1))
{
	if(input[i]==(663+offset1))
{
	vR[i] = 663;
}
else
{
	vR[i] = 662;
}

}
else
{
	if(input[i]==(661+offset1))
{
	vR[i] = 661;
}
else
{
	vR[i] = 660;
}

}

}
else
{
	if(input[i]>(657+offset1))
{
	if(input[i]==(659+offset1))
{
	vR[i] = 659;
}
else
{
	vR[i] = 658;
}

}
else
{
	if(input[i]==(657+offset1))
{
	vR[i] = 657;
}
else
{
	vR[i] = 656;
}

}

}

}

}
else
{
	if(input[i]>(647+offset1))
{
	if(input[i]>(651+offset1))
{
	if(input[i]>(653+offset1))
{
	if(input[i]==(655+offset1))
{
	vR[i] = 655;
}
else
{
	vR[i] = 654;
}

}
else
{
	if(input[i]==(653+offset1))
{
	vR[i] = 653;
}
else
{
	vR[i] = 652;
}

}

}
else
{
	if(input[i]>(649+offset1))
{
	if(input[i]==(651+offset1))
{
	vR[i] = 651;
}
else
{
	vR[i] = 650;
}

}
else
{
	if(input[i]==(649+offset1))
{
	vR[i] = 649;
}
else
{
	vR[i] = 648;
}

}

}

}
else
{
	if(input[i]>(643+offset1))
{
	if(input[i]>(645+offset1))
{
	if(input[i]==(647+offset1))
{
	vR[i] = 647;
}
else
{
	vR[i] = 646;
}

}
else
{
	if(input[i]==(645+offset1))
{
	vR[i] = 645;
}
else
{
	vR[i] = 644;
}

}

}
else
{
	if(input[i]>(641+offset1))
{
	if(input[i]==(643+offset1))
{
	vR[i] = 643;
}
else
{
	vR[i] = 642;
}

}
else
{
	if(input[i]==(641+offset1))
{
	vR[i] = 641;
}
else
{
	vR[i] = 640;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(575+offset1))
{
	if(input[i]>(607+offset1))
{
	if(input[i]>(623+offset1))
{
	if(input[i]>(631+offset1))
{
	if(input[i]>(635+offset1))
{
	if(input[i]>(637+offset1))
{
	if(input[i]==(639+offset1))
{
	vR[i] = 639;
}
else
{
	vR[i] = 638;
}

}
else
{
	if(input[i]==(637+offset1))
{
	vR[i] = 637;
}
else
{
	vR[i] = 636;
}

}

}
else
{
	if(input[i]>(633+offset1))
{
	if(input[i]==(635+offset1))
{
	vR[i] = 635;
}
else
{
	vR[i] = 634;
}

}
else
{
	if(input[i]==(633+offset1))
{
	vR[i] = 633;
}
else
{
	vR[i] = 632;
}

}

}

}
else
{
	if(input[i]>(627+offset1))
{
	if(input[i]>(629+offset1))
{
	if(input[i]==(631+offset1))
{
	vR[i] = 631;
}
else
{
	vR[i] = 630;
}

}
else
{
	if(input[i]==(629+offset1))
{
	vR[i] = 629;
}
else
{
	vR[i] = 628;
}

}

}
else
{
	if(input[i]>(625+offset1))
{
	if(input[i]==(627+offset1))
{
	vR[i] = 627;
}
else
{
	vR[i] = 626;
}

}
else
{
	if(input[i]==(625+offset1))
{
	vR[i] = 625;
}
else
{
	vR[i] = 624;
}

}

}

}

}
else
{
	if(input[i]>(615+offset1))
{
	if(input[i]>(619+offset1))
{
	if(input[i]>(621+offset1))
{
	if(input[i]==(623+offset1))
{
	vR[i] = 623;
}
else
{
	vR[i] = 622;
}

}
else
{
	if(input[i]==(621+offset1))
{
	vR[i] = 621;
}
else
{
	vR[i] = 620;
}

}

}
else
{
	if(input[i]>(617+offset1))
{
	if(input[i]==(619+offset1))
{
	vR[i] = 619;
}
else
{
	vR[i] = 618;
}

}
else
{
	if(input[i]==(617+offset1))
{
	vR[i] = 617;
}
else
{
	vR[i] = 616;
}

}

}

}
else
{
	if(input[i]>(611+offset1))
{
	if(input[i]>(613+offset1))
{
	if(input[i]==(615+offset1))
{
	vR[i] = 615;
}
else
{
	vR[i] = 614;
}

}
else
{
	if(input[i]==(613+offset1))
{
	vR[i] = 613;
}
else
{
	vR[i] = 612;
}

}

}
else
{
	if(input[i]>(609+offset1))
{
	if(input[i]==(611+offset1))
{
	vR[i] = 611;
}
else
{
	vR[i] = 610;
}

}
else
{
	if(input[i]==(609+offset1))
{
	vR[i] = 609;
}
else
{
	vR[i] = 608;
}

}

}

}

}

}
else
{
	if(input[i]>(591+offset1))
{
	if(input[i]>(599+offset1))
{
	if(input[i]>(603+offset1))
{
	if(input[i]>(605+offset1))
{
	if(input[i]==(607+offset1))
{
	vR[i] = 607;
}
else
{
	vR[i] = 606;
}

}
else
{
	if(input[i]==(605+offset1))
{
	vR[i] = 605;
}
else
{
	vR[i] = 604;
}

}

}
else
{
	if(input[i]>(601+offset1))
{
	if(input[i]==(603+offset1))
{
	vR[i] = 603;
}
else
{
	vR[i] = 602;
}

}
else
{
	if(input[i]==(601+offset1))
{
	vR[i] = 601;
}
else
{
	vR[i] = 600;
}

}

}

}
else
{
	if(input[i]>(595+offset1))
{
	if(input[i]>(597+offset1))
{
	if(input[i]==(599+offset1))
{
	vR[i] = 599;
}
else
{
	vR[i] = 598;
}

}
else
{
	if(input[i]==(597+offset1))
{
	vR[i] = 597;
}
else
{
	vR[i] = 596;
}

}

}
else
{
	if(input[i]>(593+offset1))
{
	if(input[i]==(595+offset1))
{
	vR[i] = 595;
}
else
{
	vR[i] = 594;
}

}
else
{
	if(input[i]==(593+offset1))
{
	vR[i] = 593;
}
else
{
	vR[i] = 592;
}

}

}

}

}
else
{
	if(input[i]>(583+offset1))
{
	if(input[i]>(587+offset1))
{
	if(input[i]>(589+offset1))
{
	if(input[i]==(591+offset1))
{
	vR[i] = 591;
}
else
{
	vR[i] = 590;
}

}
else
{
	if(input[i]==(589+offset1))
{
	vR[i] = 589;
}
else
{
	vR[i] = 588;
}

}

}
else
{
	if(input[i]>(585+offset1))
{
	if(input[i]==(587+offset1))
{
	vR[i] = 587;
}
else
{
	vR[i] = 586;
}

}
else
{
	if(input[i]==(585+offset1))
{
	vR[i] = 585;
}
else
{
	vR[i] = 584;
}

}

}

}
else
{
	if(input[i]>(579+offset1))
{
	if(input[i]>(581+offset1))
{
	if(input[i]==(583+offset1))
{
	vR[i] = 583;
}
else
{
	vR[i] = 582;
}

}
else
{
	if(input[i]==(581+offset1))
{
	vR[i] = 581;
}
else
{
	vR[i] = 580;
}

}

}
else
{
	if(input[i]>(577+offset1))
{
	if(input[i]==(579+offset1))
{
	vR[i] = 579;
}
else
{
	vR[i] = 578;
}

}
else
{
	if(input[i]==(577+offset1))
{
	vR[i] = 577;
}
else
{
	vR[i] = 576;
}

}

}

}

}

}

}
else
{
	if(input[i]>(543+offset1))
{
	if(input[i]>(559+offset1))
{
	if(input[i]>(567+offset1))
{
	if(input[i]>(571+offset1))
{
	if(input[i]>(573+offset1))
{
	if(input[i]==(575+offset1))
{
	vR[i] = 575;
}
else
{
	vR[i] = 574;
}

}
else
{
	if(input[i]==(573+offset1))
{
	vR[i] = 573;
}
else
{
	vR[i] = 572;
}

}

}
else
{
	if(input[i]>(569+offset1))
{
	if(input[i]==(571+offset1))
{
	vR[i] = 571;
}
else
{
	vR[i] = 570;
}

}
else
{
	if(input[i]==(569+offset1))
{
	vR[i] = 569;
}
else
{
	vR[i] = 568;
}

}

}

}
else
{
	if(input[i]>(563+offset1))
{
	if(input[i]>(565+offset1))
{
	if(input[i]==(567+offset1))
{
	vR[i] = 567;
}
else
{
	vR[i] = 566;
}

}
else
{
	if(input[i]==(565+offset1))
{
	vR[i] = 565;
}
else
{
	vR[i] = 564;
}

}

}
else
{
	if(input[i]>(561+offset1))
{
	if(input[i]==(563+offset1))
{
	vR[i] = 563;
}
else
{
	vR[i] = 562;
}

}
else
{
	if(input[i]==(561+offset1))
{
	vR[i] = 561;
}
else
{
	vR[i] = 560;
}

}

}

}

}
else
{
	if(input[i]>(551+offset1))
{
	if(input[i]>(555+offset1))
{
	if(input[i]>(557+offset1))
{
	if(input[i]==(559+offset1))
{
	vR[i] = 559;
}
else
{
	vR[i] = 558;
}

}
else
{
	if(input[i]==(557+offset1))
{
	vR[i] = 557;
}
else
{
	vR[i] = 556;
}

}

}
else
{
	if(input[i]>(553+offset1))
{
	if(input[i]==(555+offset1))
{
	vR[i] = 555;
}
else
{
	vR[i] = 554;
}

}
else
{
	if(input[i]==(553+offset1))
{
	vR[i] = 553;
}
else
{
	vR[i] = 552;
}

}

}

}
else
{
	if(input[i]>(547+offset1))
{
	if(input[i]>(549+offset1))
{
	if(input[i]==(551+offset1))
{
	vR[i] = 551;
}
else
{
	vR[i] = 550;
}

}
else
{
	if(input[i]==(549+offset1))
{
	vR[i] = 549;
}
else
{
	vR[i] = 548;
}

}

}
else
{
	if(input[i]>(545+offset1))
{
	if(input[i]==(547+offset1))
{
	vR[i] = 547;
}
else
{
	vR[i] = 546;
}

}
else
{
	if(input[i]==(545+offset1))
{
	vR[i] = 545;
}
else
{
	vR[i] = 544;
}

}

}

}

}

}
else
{
	if(input[i]>(527+offset1))
{
	if(input[i]>(535+offset1))
{
	if(input[i]>(539+offset1))
{
	if(input[i]>(541+offset1))
{
	if(input[i]==(543+offset1))
{
	vR[i] = 543;
}
else
{
	vR[i] = 542;
}

}
else
{
	if(input[i]==(541+offset1))
{
	vR[i] = 541;
}
else
{
	vR[i] = 540;
}

}

}
else
{
	if(input[i]>(537+offset1))
{
	if(input[i]==(539+offset1))
{
	vR[i] = 539;
}
else
{
	vR[i] = 538;
}

}
else
{
	if(input[i]==(537+offset1))
{
	vR[i] = 537;
}
else
{
	vR[i] = 536;
}

}

}

}
else
{
	if(input[i]>(531+offset1))
{
	if(input[i]>(533+offset1))
{
	if(input[i]==(535+offset1))
{
	vR[i] = 535;
}
else
{
	vR[i] = 534;
}

}
else
{
	if(input[i]==(533+offset1))
{
	vR[i] = 533;
}
else
{
	vR[i] = 532;
}

}

}
else
{
	if(input[i]>(529+offset1))
{
	if(input[i]==(531+offset1))
{
	vR[i] = 531;
}
else
{
	vR[i] = 530;
}

}
else
{
	if(input[i]==(529+offset1))
{
	vR[i] = 529;
}
else
{
	vR[i] = 528;
}

}

}

}

}
else
{
	if(input[i]>(519+offset1))
{
	if(input[i]>(523+offset1))
{
	if(input[i]>(525+offset1))
{
	if(input[i]==(527+offset1))
{
	vR[i] = 527;
}
else
{
	vR[i] = 526;
}

}
else
{
	if(input[i]==(525+offset1))
{
	vR[i] = 525;
}
else
{
	vR[i] = 524;
}

}

}
else
{
	if(input[i]>(521+offset1))
{
	if(input[i]==(523+offset1))
{
	vR[i] = 523;
}
else
{
	vR[i] = 522;
}

}
else
{
	if(input[i]==(521+offset1))
{
	vR[i] = 521;
}
else
{
	vR[i] = 520;
}

}

}

}
else
{
	if(input[i]>(515+offset1))
{
	if(input[i]>(517+offset1))
{
	if(input[i]==(519+offset1))
{
	vR[i] = 519;
}
else
{
	vR[i] = 518;
}

}
else
{
	if(input[i]==(517+offset1))
{
	vR[i] = 517;
}
else
{
	vR[i] = 516;
}

}

}
else
{
	if(input[i]>(513+offset1))
{
	if(input[i]==(515+offset1))
{
	vR[i] = 515;
}
else
{
	vR[i] = 514;
}

}
else
{
	if(input[i]==(513+offset1))
{
	vR[i] = 513;
}
else
{
	vR[i] = 512;
}

}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(255+offset1))
{
	if(input[i]>(383+offset1))
{
	if(input[i]>(447+offset1))
{
	if(input[i]>(479+offset1))
{
	if(input[i]>(495+offset1))
{
	if(input[i]>(503+offset1))
{
	if(input[i]>(507+offset1))
{
	if(input[i]>(509+offset1))
{
	if(input[i]==(511+offset1))
{
	vR[i] = 511;
}
else
{
	vR[i] = 510;
}

}
else
{
	if(input[i]==(509+offset1))
{
	vR[i] = 509;
}
else
{
	vR[i] = 508;
}

}

}
else
{
	if(input[i]>(505+offset1))
{
	if(input[i]==(507+offset1))
{
	vR[i] = 507;
}
else
{
	vR[i] = 506;
}

}
else
{
	if(input[i]==(505+offset1))
{
	vR[i] = 505;
}
else
{
	vR[i] = 504;
}

}

}

}
else
{
	if(input[i]>(499+offset1))
{
	if(input[i]>(501+offset1))
{
	if(input[i]==(503+offset1))
{
	vR[i] = 503;
}
else
{
	vR[i] = 502;
}

}
else
{
	if(input[i]==(501+offset1))
{
	vR[i] = 501;
}
else
{
	vR[i] = 500;
}

}

}
else
{
	if(input[i]>(497+offset1))
{
	if(input[i]==(499+offset1))
{
	vR[i] = 499;
}
else
{
	vR[i] = 498;
}

}
else
{
	if(input[i]==(497+offset1))
{
	vR[i] = 497;
}
else
{
	vR[i] = 496;
}

}

}

}

}
else
{
	if(input[i]>(487+offset1))
{
	if(input[i]>(491+offset1))
{
	if(input[i]>(493+offset1))
{
	if(input[i]==(495+offset1))
{
	vR[i] = 495;
}
else
{
	vR[i] = 494;
}

}
else
{
	if(input[i]==(493+offset1))
{
	vR[i] = 493;
}
else
{
	vR[i] = 492;
}

}

}
else
{
	if(input[i]>(489+offset1))
{
	if(input[i]==(491+offset1))
{
	vR[i] = 491;
}
else
{
	vR[i] = 490;
}

}
else
{
	if(input[i]==(489+offset1))
{
	vR[i] = 489;
}
else
{
	vR[i] = 488;
}

}

}

}
else
{
	if(input[i]>(483+offset1))
{
	if(input[i]>(485+offset1))
{
	if(input[i]==(487+offset1))
{
	vR[i] = 487;
}
else
{
	vR[i] = 486;
}

}
else
{
	if(input[i]==(485+offset1))
{
	vR[i] = 485;
}
else
{
	vR[i] = 484;
}

}

}
else
{
	if(input[i]>(481+offset1))
{
	if(input[i]==(483+offset1))
{
	vR[i] = 483;
}
else
{
	vR[i] = 482;
}

}
else
{
	if(input[i]==(481+offset1))
{
	vR[i] = 481;
}
else
{
	vR[i] = 480;
}

}

}

}

}

}
else
{
	if(input[i]>(463+offset1))
{
	if(input[i]>(471+offset1))
{
	if(input[i]>(475+offset1))
{
	if(input[i]>(477+offset1))
{
	if(input[i]==(479+offset1))
{
	vR[i] = 479;
}
else
{
	vR[i] = 478;
}

}
else
{
	if(input[i]==(477+offset1))
{
	vR[i] = 477;
}
else
{
	vR[i] = 476;
}

}

}
else
{
	if(input[i]>(473+offset1))
{
	if(input[i]==(475+offset1))
{
	vR[i] = 475;
}
else
{
	vR[i] = 474;
}

}
else
{
	if(input[i]==(473+offset1))
{
	vR[i] = 473;
}
else
{
	vR[i] = 472;
}

}

}

}
else
{
	if(input[i]>(467+offset1))
{
	if(input[i]>(469+offset1))
{
	if(input[i]==(471+offset1))
{
	vR[i] = 471;
}
else
{
	vR[i] = 470;
}

}
else
{
	if(input[i]==(469+offset1))
{
	vR[i] = 469;
}
else
{
	vR[i] = 468;
}

}

}
else
{
	if(input[i]>(465+offset1))
{
	if(input[i]==(467+offset1))
{
	vR[i] = 467;
}
else
{
	vR[i] = 466;
}

}
else
{
	if(input[i]==(465+offset1))
{
	vR[i] = 465;
}
else
{
	vR[i] = 464;
}

}

}

}

}
else
{
	if(input[i]>(455+offset1))
{
	if(input[i]>(459+offset1))
{
	if(input[i]>(461+offset1))
{
	if(input[i]==(463+offset1))
{
	vR[i] = 463;
}
else
{
	vR[i] = 462;
}

}
else
{
	if(input[i]==(461+offset1))
{
	vR[i] = 461;
}
else
{
	vR[i] = 460;
}

}

}
else
{
	if(input[i]>(457+offset1))
{
	if(input[i]==(459+offset1))
{
	vR[i] = 459;
}
else
{
	vR[i] = 458;
}

}
else
{
	if(input[i]==(457+offset1))
{
	vR[i] = 457;
}
else
{
	vR[i] = 456;
}

}

}

}
else
{
	if(input[i]>(451+offset1))
{
	if(input[i]>(453+offset1))
{
	if(input[i]==(455+offset1))
{
	vR[i] = 455;
}
else
{
	vR[i] = 454;
}

}
else
{
	if(input[i]==(453+offset1))
{
	vR[i] = 453;
}
else
{
	vR[i] = 452;
}

}

}
else
{
	if(input[i]>(449+offset1))
{
	if(input[i]==(451+offset1))
{
	vR[i] = 451;
}
else
{
	vR[i] = 450;
}

}
else
{
	if(input[i]==(449+offset1))
{
	vR[i] = 449;
}
else
{
	vR[i] = 448;
}

}

}

}

}

}

}
else
{
	if(input[i]>(415+offset1))
{
	if(input[i]>(431+offset1))
{
	if(input[i]>(439+offset1))
{
	if(input[i]>(443+offset1))
{
	if(input[i]>(445+offset1))
{
	if(input[i]==(447+offset1))
{
	vR[i] = 447;
}
else
{
	vR[i] = 446;
}

}
else
{
	if(input[i]==(445+offset1))
{
	vR[i] = 445;
}
else
{
	vR[i] = 444;
}

}

}
else
{
	if(input[i]>(441+offset1))
{
	if(input[i]==(443+offset1))
{
	vR[i] = 443;
}
else
{
	vR[i] = 442;
}

}
else
{
	if(input[i]==(441+offset1))
{
	vR[i] = 441;
}
else
{
	vR[i] = 440;
}

}

}

}
else
{
	if(input[i]>(435+offset1))
{
	if(input[i]>(437+offset1))
{
	if(input[i]==(439+offset1))
{
	vR[i] = 439;
}
else
{
	vR[i] = 438;
}

}
else
{
	if(input[i]==(437+offset1))
{
	vR[i] = 437;
}
else
{
	vR[i] = 436;
}

}

}
else
{
	if(input[i]>(433+offset1))
{
	if(input[i]==(435+offset1))
{
	vR[i] = 435;
}
else
{
	vR[i] = 434;
}

}
else
{
	if(input[i]==(433+offset1))
{
	vR[i] = 433;
}
else
{
	vR[i] = 432;
}

}

}

}

}
else
{
	if(input[i]>(423+offset1))
{
	if(input[i]>(427+offset1))
{
	if(input[i]>(429+offset1))
{
	if(input[i]==(431+offset1))
{
	vR[i] = 431;
}
else
{
	vR[i] = 430;
}

}
else
{
	if(input[i]==(429+offset1))
{
	vR[i] = 429;
}
else
{
	vR[i] = 428;
}

}

}
else
{
	if(input[i]>(425+offset1))
{
	if(input[i]==(427+offset1))
{
	vR[i] = 427;
}
else
{
	vR[i] = 426;
}

}
else
{
	if(input[i]==(425+offset1))
{
	vR[i] = 425;
}
else
{
	vR[i] = 424;
}

}

}

}
else
{
	if(input[i]>(419+offset1))
{
	if(input[i]>(421+offset1))
{
	if(input[i]==(423+offset1))
{
	vR[i] = 423;
}
else
{
	vR[i] = 422;
}

}
else
{
	if(input[i]==(421+offset1))
{
	vR[i] = 421;
}
else
{
	vR[i] = 420;
}

}

}
else
{
	if(input[i]>(417+offset1))
{
	if(input[i]==(419+offset1))
{
	vR[i] = 419;
}
else
{
	vR[i] = 418;
}

}
else
{
	if(input[i]==(417+offset1))
{
	vR[i] = 417;
}
else
{
	vR[i] = 416;
}

}

}

}

}

}
else
{
	if(input[i]>(399+offset1))
{
	if(input[i]>(407+offset1))
{
	if(input[i]>(411+offset1))
{
	if(input[i]>(413+offset1))
{
	if(input[i]==(415+offset1))
{
	vR[i] = 415;
}
else
{
	vR[i] = 414;
}

}
else
{
	if(input[i]==(413+offset1))
{
	vR[i] = 413;
}
else
{
	vR[i] = 412;
}

}

}
else
{
	if(input[i]>(409+offset1))
{
	if(input[i]==(411+offset1))
{
	vR[i] = 411;
}
else
{
	vR[i] = 410;
}

}
else
{
	if(input[i]==(409+offset1))
{
	vR[i] = 409;
}
else
{
	vR[i] = 408;
}

}

}

}
else
{
	if(input[i]>(403+offset1))
{
	if(input[i]>(405+offset1))
{
	if(input[i]==(407+offset1))
{
	vR[i] = 407;
}
else
{
	vR[i] = 406;
}

}
else
{
	if(input[i]==(405+offset1))
{
	vR[i] = 405;
}
else
{
	vR[i] = 404;
}

}

}
else
{
	if(input[i]>(401+offset1))
{
	if(input[i]==(403+offset1))
{
	vR[i] = 403;
}
else
{
	vR[i] = 402;
}

}
else
{
	if(input[i]==(401+offset1))
{
	vR[i] = 401;
}
else
{
	vR[i] = 400;
}

}

}

}

}
else
{
	if(input[i]>(391+offset1))
{
	if(input[i]>(395+offset1))
{
	if(input[i]>(397+offset1))
{
	if(input[i]==(399+offset1))
{
	vR[i] = 399;
}
else
{
	vR[i] = 398;
}

}
else
{
	if(input[i]==(397+offset1))
{
	vR[i] = 397;
}
else
{
	vR[i] = 396;
}

}

}
else
{
	if(input[i]>(393+offset1))
{
	if(input[i]==(395+offset1))
{
	vR[i] = 395;
}
else
{
	vR[i] = 394;
}

}
else
{
	if(input[i]==(393+offset1))
{
	vR[i] = 393;
}
else
{
	vR[i] = 392;
}

}

}

}
else
{
	if(input[i]>(387+offset1))
{
	if(input[i]>(389+offset1))
{
	if(input[i]==(391+offset1))
{
	vR[i] = 391;
}
else
{
	vR[i] = 390;
}

}
else
{
	if(input[i]==(389+offset1))
{
	vR[i] = 389;
}
else
{
	vR[i] = 388;
}

}

}
else
{
	if(input[i]>(385+offset1))
{
	if(input[i]==(387+offset1))
{
	vR[i] = 387;
}
else
{
	vR[i] = 386;
}

}
else
{
	if(input[i]==(385+offset1))
{
	vR[i] = 385;
}
else
{
	vR[i] = 384;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(319+offset1))
{
	if(input[i]>(351+offset1))
{
	if(input[i]>(367+offset1))
{
	if(input[i]>(375+offset1))
{
	if(input[i]>(379+offset1))
{
	if(input[i]>(381+offset1))
{
	if(input[i]==(383+offset1))
{
	vR[i] = 383;
}
else
{
	vR[i] = 382;
}

}
else
{
	if(input[i]==(381+offset1))
{
	vR[i] = 381;
}
else
{
	vR[i] = 380;
}

}

}
else
{
	if(input[i]>(377+offset1))
{
	if(input[i]==(379+offset1))
{
	vR[i] = 379;
}
else
{
	vR[i] = 378;
}

}
else
{
	if(input[i]==(377+offset1))
{
	vR[i] = 377;
}
else
{
	vR[i] = 376;
}

}

}

}
else
{
	if(input[i]>(371+offset1))
{
	if(input[i]>(373+offset1))
{
	if(input[i]==(375+offset1))
{
	vR[i] = 375;
}
else
{
	vR[i] = 374;
}

}
else
{
	if(input[i]==(373+offset1))
{
	vR[i] = 373;
}
else
{
	vR[i] = 372;
}

}

}
else
{
	if(input[i]>(369+offset1))
{
	if(input[i]==(371+offset1))
{
	vR[i] = 371;
}
else
{
	vR[i] = 370;
}

}
else
{
	if(input[i]==(369+offset1))
{
	vR[i] = 369;
}
else
{
	vR[i] = 368;
}

}

}

}

}
else
{
	if(input[i]>(359+offset1))
{
	if(input[i]>(363+offset1))
{
	if(input[i]>(365+offset1))
{
	if(input[i]==(367+offset1))
{
	vR[i] = 367;
}
else
{
	vR[i] = 366;
}

}
else
{
	if(input[i]==(365+offset1))
{
	vR[i] = 365;
}
else
{
	vR[i] = 364;
}

}

}
else
{
	if(input[i]>(361+offset1))
{
	if(input[i]==(363+offset1))
{
	vR[i] = 363;
}
else
{
	vR[i] = 362;
}

}
else
{
	if(input[i]==(361+offset1))
{
	vR[i] = 361;
}
else
{
	vR[i] = 360;
}

}

}

}
else
{
	if(input[i]>(355+offset1))
{
	if(input[i]>(357+offset1))
{
	if(input[i]==(359+offset1))
{
	vR[i] = 359;
}
else
{
	vR[i] = 358;
}

}
else
{
	if(input[i]==(357+offset1))
{
	vR[i] = 357;
}
else
{
	vR[i] = 356;
}

}

}
else
{
	if(input[i]>(353+offset1))
{
	if(input[i]==(355+offset1))
{
	vR[i] = 355;
}
else
{
	vR[i] = 354;
}

}
else
{
	if(input[i]==(353+offset1))
{
	vR[i] = 353;
}
else
{
	vR[i] = 352;
}

}

}

}

}

}
else
{
	if(input[i]>(335+offset1))
{
	if(input[i]>(343+offset1))
{
	if(input[i]>(347+offset1))
{
	if(input[i]>(349+offset1))
{
	if(input[i]==(351+offset1))
{
	vR[i] = 351;
}
else
{
	vR[i] = 350;
}

}
else
{
	if(input[i]==(349+offset1))
{
	vR[i] = 349;
}
else
{
	vR[i] = 348;
}

}

}
else
{
	if(input[i]>(345+offset1))
{
	if(input[i]==(347+offset1))
{
	vR[i] = 347;
}
else
{
	vR[i] = 346;
}

}
else
{
	if(input[i]==(345+offset1))
{
	vR[i] = 345;
}
else
{
	vR[i] = 344;
}

}

}

}
else
{
	if(input[i]>(339+offset1))
{
	if(input[i]>(341+offset1))
{
	if(input[i]==(343+offset1))
{
	vR[i] = 343;
}
else
{
	vR[i] = 342;
}

}
else
{
	if(input[i]==(341+offset1))
{
	vR[i] = 341;
}
else
{
	vR[i] = 340;
}

}

}
else
{
	if(input[i]>(337+offset1))
{
	if(input[i]==(339+offset1))
{
	vR[i] = 339;
}
else
{
	vR[i] = 338;
}

}
else
{
	if(input[i]==(337+offset1))
{
	vR[i] = 337;
}
else
{
	vR[i] = 336;
}

}

}

}

}
else
{
	if(input[i]>(327+offset1))
{
	if(input[i]>(331+offset1))
{
	if(input[i]>(333+offset1))
{
	if(input[i]==(335+offset1))
{
	vR[i] = 335;
}
else
{
	vR[i] = 334;
}

}
else
{
	if(input[i]==(333+offset1))
{
	vR[i] = 333;
}
else
{
	vR[i] = 332;
}

}

}
else
{
	if(input[i]>(329+offset1))
{
	if(input[i]==(331+offset1))
{
	vR[i] = 331;
}
else
{
	vR[i] = 330;
}

}
else
{
	if(input[i]==(329+offset1))
{
	vR[i] = 329;
}
else
{
	vR[i] = 328;
}

}

}

}
else
{
	if(input[i]>(323+offset1))
{
	if(input[i]>(325+offset1))
{
	if(input[i]==(327+offset1))
{
	vR[i] = 327;
}
else
{
	vR[i] = 326;
}

}
else
{
	if(input[i]==(325+offset1))
{
	vR[i] = 325;
}
else
{
	vR[i] = 324;
}

}

}
else
{
	if(input[i]>(321+offset1))
{
	if(input[i]==(323+offset1))
{
	vR[i] = 323;
}
else
{
	vR[i] = 322;
}

}
else
{
	if(input[i]==(321+offset1))
{
	vR[i] = 321;
}
else
{
	vR[i] = 320;
}

}

}

}

}

}

}
else
{
	if(input[i]>(287+offset1))
{
	if(input[i]>(303+offset1))
{
	if(input[i]>(311+offset1))
{
	if(input[i]>(315+offset1))
{
	if(input[i]>(317+offset1))
{
	if(input[i]==(319+offset1))
{
	vR[i] = 319;
}
else
{
	vR[i] = 318;
}

}
else
{
	if(input[i]==(317+offset1))
{
	vR[i] = 317;
}
else
{
	vR[i] = 316;
}

}

}
else
{
	if(input[i]>(313+offset1))
{
	if(input[i]==(315+offset1))
{
	vR[i] = 315;
}
else
{
	vR[i] = 314;
}

}
else
{
	if(input[i]==(313+offset1))
{
	vR[i] = 313;
}
else
{
	vR[i] = 312;
}

}

}

}
else
{
	if(input[i]>(307+offset1))
{
	if(input[i]>(309+offset1))
{
	if(input[i]==(311+offset1))
{
	vR[i] = 311;
}
else
{
	vR[i] = 310;
}

}
else
{
	if(input[i]==(309+offset1))
{
	vR[i] = 309;
}
else
{
	vR[i] = 308;
}

}

}
else
{
	if(input[i]>(305+offset1))
{
	if(input[i]==(307+offset1))
{
	vR[i] = 307;
}
else
{
	vR[i] = 306;
}

}
else
{
	if(input[i]==(305+offset1))
{
	vR[i] = 305;
}
else
{
	vR[i] = 304;
}

}

}

}

}
else
{
	if(input[i]>(295+offset1))
{
	if(input[i]>(299+offset1))
{
	if(input[i]>(301+offset1))
{
	if(input[i]==(303+offset1))
{
	vR[i] = 303;
}
else
{
	vR[i] = 302;
}

}
else
{
	if(input[i]==(301+offset1))
{
	vR[i] = 301;
}
else
{
	vR[i] = 300;
}

}

}
else
{
	if(input[i]>(297+offset1))
{
	if(input[i]==(299+offset1))
{
	vR[i] = 299;
}
else
{
	vR[i] = 298;
}

}
else
{
	if(input[i]==(297+offset1))
{
	vR[i] = 297;
}
else
{
	vR[i] = 296;
}

}

}

}
else
{
	if(input[i]>(291+offset1))
{
	if(input[i]>(293+offset1))
{
	if(input[i]==(295+offset1))
{
	vR[i] = 295;
}
else
{
	vR[i] = 294;
}

}
else
{
	if(input[i]==(293+offset1))
{
	vR[i] = 293;
}
else
{
	vR[i] = 292;
}

}

}
else
{
	if(input[i]>(289+offset1))
{
	if(input[i]==(291+offset1))
{
	vR[i] = 291;
}
else
{
	vR[i] = 290;
}

}
else
{
	if(input[i]==(289+offset1))
{
	vR[i] = 289;
}
else
{
	vR[i] = 288;
}

}

}

}

}

}
else
{
	if(input[i]>(271+offset1))
{
	if(input[i]>(279+offset1))
{
	if(input[i]>(283+offset1))
{
	if(input[i]>(285+offset1))
{
	if(input[i]==(287+offset1))
{
	vR[i] = 287;
}
else
{
	vR[i] = 286;
}

}
else
{
	if(input[i]==(285+offset1))
{
	vR[i] = 285;
}
else
{
	vR[i] = 284;
}

}

}
else
{
	if(input[i]>(281+offset1))
{
	if(input[i]==(283+offset1))
{
	vR[i] = 283;
}
else
{
	vR[i] = 282;
}

}
else
{
	if(input[i]==(281+offset1))
{
	vR[i] = 281;
}
else
{
	vR[i] = 280;
}

}

}

}
else
{
	if(input[i]>(275+offset1))
{
	if(input[i]>(277+offset1))
{
	if(input[i]==(279+offset1))
{
	vR[i] = 279;
}
else
{
	vR[i] = 278;
}

}
else
{
	if(input[i]==(277+offset1))
{
	vR[i] = 277;
}
else
{
	vR[i] = 276;
}

}

}
else
{
	if(input[i]>(273+offset1))
{
	if(input[i]==(275+offset1))
{
	vR[i] = 275;
}
else
{
	vR[i] = 274;
}

}
else
{
	if(input[i]==(273+offset1))
{
	vR[i] = 273;
}
else
{
	vR[i] = 272;
}

}

}

}

}
else
{
	if(input[i]>(263+offset1))
{
	if(input[i]>(267+offset1))
{
	if(input[i]>(269+offset1))
{
	if(input[i]==(271+offset1))
{
	vR[i] = 271;
}
else
{
	vR[i] = 270;
}

}
else
{
	if(input[i]==(269+offset1))
{
	vR[i] = 269;
}
else
{
	vR[i] = 268;
}

}

}
else
{
	if(input[i]>(265+offset1))
{
	if(input[i]==(267+offset1))
{
	vR[i] = 267;
}
else
{
	vR[i] = 266;
}

}
else
{
	if(input[i]==(265+offset1))
{
	vR[i] = 265;
}
else
{
	vR[i] = 264;
}

}

}

}
else
{
	if(input[i]>(259+offset1))
{
	if(input[i]>(261+offset1))
{
	if(input[i]==(263+offset1))
{
	vR[i] = 263;
}
else
{
	vR[i] = 262;
}

}
else
{
	if(input[i]==(261+offset1))
{
	vR[i] = 261;
}
else
{
	vR[i] = 260;
}

}

}
else
{
	if(input[i]>(257+offset1))
{
	if(input[i]==(259+offset1))
{
	vR[i] = 259;
}
else
{
	vR[i] = 258;
}

}
else
{
	if(input[i]==(257+offset1))
{
	vR[i] = 257;
}
else
{
	vR[i] = 256;
}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(127+offset1))
{
	if(input[i]>(191+offset1))
{
	if(input[i]>(223+offset1))
{
	if(input[i]>(239+offset1))
{
	if(input[i]>(247+offset1))
{
	if(input[i]>(251+offset1))
{
	if(input[i]>(253+offset1))
{
	if(input[i]==(255+offset1))
{
	vR[i] = 255;
}
else
{
	vR[i] = 254;
}

}
else
{
	if(input[i]==(253+offset1))
{
	vR[i] = 253;
}
else
{
	vR[i] = 252;
}

}

}
else
{
	if(input[i]>(249+offset1))
{
	if(input[i]==(251+offset1))
{
	vR[i] = 251;
}
else
{
	vR[i] = 250;
}

}
else
{
	if(input[i]==(249+offset1))
{
	vR[i] = 249;
}
else
{
	vR[i] = 248;
}

}

}

}
else
{
	if(input[i]>(243+offset1))
{
	if(input[i]>(245+offset1))
{
	if(input[i]==(247+offset1))
{
	vR[i] = 247;
}
else
{
	vR[i] = 246;
}

}
else
{
	if(input[i]==(245+offset1))
{
	vR[i] = 245;
}
else
{
	vR[i] = 244;
}

}

}
else
{
	if(input[i]>(241+offset1))
{
	if(input[i]==(243+offset1))
{
	vR[i] = 243;
}
else
{
	vR[i] = 242;
}

}
else
{
	if(input[i]==(241+offset1))
{
	vR[i] = 241;
}
else
{
	vR[i] = 240;
}

}

}

}

}
else
{
	if(input[i]>(231+offset1))
{
	if(input[i]>(235+offset1))
{
	if(input[i]>(237+offset1))
{
	if(input[i]==(239+offset1))
{
	vR[i] = 239;
}
else
{
	vR[i] = 238;
}

}
else
{
	if(input[i]==(237+offset1))
{
	vR[i] = 237;
}
else
{
	vR[i] = 236;
}

}

}
else
{
	if(input[i]>(233+offset1))
{
	if(input[i]==(235+offset1))
{
	vR[i] = 235;
}
else
{
	vR[i] = 234;
}

}
else
{
	if(input[i]==(233+offset1))
{
	vR[i] = 233;
}
else
{
	vR[i] = 232;
}

}

}

}
else
{
	if(input[i]>(227+offset1))
{
	if(input[i]>(229+offset1))
{
	if(input[i]==(231+offset1))
{
	vR[i] = 231;
}
else
{
	vR[i] = 230;
}

}
else
{
	if(input[i]==(229+offset1))
{
	vR[i] = 229;
}
else
{
	vR[i] = 228;
}

}

}
else
{
	if(input[i]>(225+offset1))
{
	if(input[i]==(227+offset1))
{
	vR[i] = 227;
}
else
{
	vR[i] = 226;
}

}
else
{
	if(input[i]==(225+offset1))
{
	vR[i] = 225;
}
else
{
	vR[i] = 224;
}

}

}

}

}

}
else
{
	if(input[i]>(207+offset1))
{
	if(input[i]>(215+offset1))
{
	if(input[i]>(219+offset1))
{
	if(input[i]>(221+offset1))
{
	if(input[i]==(223+offset1))
{
	vR[i] = 223;
}
else
{
	vR[i] = 222;
}

}
else
{
	if(input[i]==(221+offset1))
{
	vR[i] = 221;
}
else
{
	vR[i] = 220;
}

}

}
else
{
	if(input[i]>(217+offset1))
{
	if(input[i]==(219+offset1))
{
	vR[i] = 219;
}
else
{
	vR[i] = 218;
}

}
else
{
	if(input[i]==(217+offset1))
{
	vR[i] = 217;
}
else
{
	vR[i] = 216;
}

}

}

}
else
{
	if(input[i]>(211+offset1))
{
	if(input[i]>(213+offset1))
{
	if(input[i]==(215+offset1))
{
	vR[i] = 215;
}
else
{
	vR[i] = 214;
}

}
else
{
	if(input[i]==(213+offset1))
{
	vR[i] = 213;
}
else
{
	vR[i] = 212;
}

}

}
else
{
	if(input[i]>(209+offset1))
{
	if(input[i]==(211+offset1))
{
	vR[i] = 211;
}
else
{
	vR[i] = 210;
}

}
else
{
	if(input[i]==(209+offset1))
{
	vR[i] = 209;
}
else
{
	vR[i] = 208;
}

}

}

}

}
else
{
	if(input[i]>(199+offset1))
{
	if(input[i]>(203+offset1))
{
	if(input[i]>(205+offset1))
{
	if(input[i]==(207+offset1))
{
	vR[i] = 207;
}
else
{
	vR[i] = 206;
}

}
else
{
	if(input[i]==(205+offset1))
{
	vR[i] = 205;
}
else
{
	vR[i] = 204;
}

}

}
else
{
	if(input[i]>(201+offset1))
{
	if(input[i]==(203+offset1))
{
	vR[i] = 203;
}
else
{
	vR[i] = 202;
}

}
else
{
	if(input[i]==(201+offset1))
{
	vR[i] = 201;
}
else
{
	vR[i] = 200;
}

}

}

}
else
{
	if(input[i]>(195+offset1))
{
	if(input[i]>(197+offset1))
{
	if(input[i]==(199+offset1))
{
	vR[i] = 199;
}
else
{
	vR[i] = 198;
}

}
else
{
	if(input[i]==(197+offset1))
{
	vR[i] = 197;
}
else
{
	vR[i] = 196;
}

}

}
else
{
	if(input[i]>(193+offset1))
{
	if(input[i]==(195+offset1))
{
	vR[i] = 195;
}
else
{
	vR[i] = 194;
}

}
else
{
	if(input[i]==(193+offset1))
{
	vR[i] = 193;
}
else
{
	vR[i] = 192;
}

}

}

}

}

}

}
else
{
	if(input[i]>(159+offset1))
{
	if(input[i]>(175+offset1))
{
	if(input[i]>(183+offset1))
{
	if(input[i]>(187+offset1))
{
	if(input[i]>(189+offset1))
{
	if(input[i]==(191+offset1))
{
	vR[i] = 191;
}
else
{
	vR[i] = 190;
}

}
else
{
	if(input[i]==(189+offset1))
{
	vR[i] = 189;
}
else
{
	vR[i] = 188;
}

}

}
else
{
	if(input[i]>(185+offset1))
{
	if(input[i]==(187+offset1))
{
	vR[i] = 187;
}
else
{
	vR[i] = 186;
}

}
else
{
	if(input[i]==(185+offset1))
{
	vR[i] = 185;
}
else
{
	vR[i] = 184;
}

}

}

}
else
{
	if(input[i]>(179+offset1))
{
	if(input[i]>(181+offset1))
{
	if(input[i]==(183+offset1))
{
	vR[i] = 183;
}
else
{
	vR[i] = 182;
}

}
else
{
	if(input[i]==(181+offset1))
{
	vR[i] = 181;
}
else
{
	vR[i] = 180;
}

}

}
else
{
	if(input[i]>(177+offset1))
{
	if(input[i]==(179+offset1))
{
	vR[i] = 179;
}
else
{
	vR[i] = 178;
}

}
else
{
	if(input[i]==(177+offset1))
{
	vR[i] = 177;
}
else
{
	vR[i] = 176;
}

}

}

}

}
else
{
	if(input[i]>(167+offset1))
{
	if(input[i]>(171+offset1))
{
	if(input[i]>(173+offset1))
{
	if(input[i]==(175+offset1))
{
	vR[i] = 175;
}
else
{
	vR[i] = 174;
}

}
else
{
	if(input[i]==(173+offset1))
{
	vR[i] = 173;
}
else
{
	vR[i] = 172;
}

}

}
else
{
	if(input[i]>(169+offset1))
{
	if(input[i]==(171+offset1))
{
	vR[i] = 171;
}
else
{
	vR[i] = 170;
}

}
else
{
	if(input[i]==(169+offset1))
{
	vR[i] = 169;
}
else
{
	vR[i] = 168;
}

}

}

}
else
{
	if(input[i]>(163+offset1))
{
	if(input[i]>(165+offset1))
{
	if(input[i]==(167+offset1))
{
	vR[i] = 167;
}
else
{
	vR[i] = 166;
}

}
else
{
	if(input[i]==(165+offset1))
{
	vR[i] = 165;
}
else
{
	vR[i] = 164;
}

}

}
else
{
	if(input[i]>(161+offset1))
{
	if(input[i]==(163+offset1))
{
	vR[i] = 163;
}
else
{
	vR[i] = 162;
}

}
else
{
	if(input[i]==(161+offset1))
{
	vR[i] = 161;
}
else
{
	vR[i] = 160;
}

}

}

}

}

}
else
{
	if(input[i]>(143+offset1))
{
	if(input[i]>(151+offset1))
{
	if(input[i]>(155+offset1))
{
	if(input[i]>(157+offset1))
{
	if(input[i]==(159+offset1))
{
	vR[i] = 159;
}
else
{
	vR[i] = 158;
}

}
else
{
	if(input[i]==(157+offset1))
{
	vR[i] = 157;
}
else
{
	vR[i] = 156;
}

}

}
else
{
	if(input[i]>(153+offset1))
{
	if(input[i]==(155+offset1))
{
	vR[i] = 155;
}
else
{
	vR[i] = 154;
}

}
else
{
	if(input[i]==(153+offset1))
{
	vR[i] = 153;
}
else
{
	vR[i] = 152;
}

}

}

}
else
{
	if(input[i]>(147+offset1))
{
	if(input[i]>(149+offset1))
{
	if(input[i]==(151+offset1))
{
	vR[i] = 151;
}
else
{
	vR[i] = 150;
}

}
else
{
	if(input[i]==(149+offset1))
{
	vR[i] = 149;
}
else
{
	vR[i] = 148;
}

}

}
else
{
	if(input[i]>(145+offset1))
{
	if(input[i]==(147+offset1))
{
	vR[i] = 147;
}
else
{
	vR[i] = 146;
}

}
else
{
	if(input[i]==(145+offset1))
{
	vR[i] = 145;
}
else
{
	vR[i] = 144;
}

}

}

}

}
else
{
	if(input[i]>(135+offset1))
{
	if(input[i]>(139+offset1))
{
	if(input[i]>(141+offset1))
{
	if(input[i]==(143+offset1))
{
	vR[i] = 143;
}
else
{
	vR[i] = 142;
}

}
else
{
	if(input[i]==(141+offset1))
{
	vR[i] = 141;
}
else
{
	vR[i] = 140;
}

}

}
else
{
	if(input[i]>(137+offset1))
{
	if(input[i]==(139+offset1))
{
	vR[i] = 139;
}
else
{
	vR[i] = 138;
}

}
else
{
	if(input[i]==(137+offset1))
{
	vR[i] = 137;
}
else
{
	vR[i] = 136;
}

}

}

}
else
{
	if(input[i]>(131+offset1))
{
	if(input[i]>(133+offset1))
{
	if(input[i]==(135+offset1))
{
	vR[i] = 135;
}
else
{
	vR[i] = 134;
}

}
else
{
	if(input[i]==(133+offset1))
{
	vR[i] = 133;
}
else
{
	vR[i] = 132;
}

}

}
else
{
	if(input[i]>(129+offset1))
{
	if(input[i]==(131+offset1))
{
	vR[i] = 131;
}
else
{
	vR[i] = 130;
}

}
else
{
	if(input[i]==(129+offset1))
{
	vR[i] = 129;
}
else
{
	vR[i] = 128;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(63+offset1))
{
	if(input[i]>(95+offset1))
{
	if(input[i]>(111+offset1))
{
	if(input[i]>(119+offset1))
{
	if(input[i]>(123+offset1))
{
	if(input[i]>(125+offset1))
{
	if(input[i]==(127+offset1))
{
	vR[i] = 127;
}
else
{
	vR[i] = 126;
}

}
else
{
	if(input[i]==(125+offset1))
{
	vR[i] = 125;
}
else
{
	vR[i] = 124;
}

}

}
else
{
	if(input[i]>(121+offset1))
{
	if(input[i]==(123+offset1))
{
	vR[i] = 123;
}
else
{
	vR[i] = 122;
}

}
else
{
	if(input[i]==(121+offset1))
{
	vR[i] = 121;
}
else
{
	vR[i] = 120;
}

}

}

}
else
{
	if(input[i]>(115+offset1))
{
	if(input[i]>(117+offset1))
{
	if(input[i]==(119+offset1))
{
	vR[i] = 119;
}
else
{
	vR[i] = 118;
}

}
else
{
	if(input[i]==(117+offset1))
{
	vR[i] = 117;
}
else
{
	vR[i] = 116;
}

}

}
else
{
	if(input[i]>(113+offset1))
{
	if(input[i]==(115+offset1))
{
	vR[i] = 115;
}
else
{
	vR[i] = 114;
}

}
else
{
	if(input[i]==(113+offset1))
{
	vR[i] = 113;
}
else
{
	vR[i] = 112;
}

}

}

}

}
else
{
	if(input[i]>(103+offset1))
{
	if(input[i]>(107+offset1))
{
	if(input[i]>(109+offset1))
{
	if(input[i]==(111+offset1))
{
	vR[i] = 111;
}
else
{
	vR[i] = 110;
}

}
else
{
	if(input[i]==(109+offset1))
{
	vR[i] = 109;
}
else
{
	vR[i] = 108;
}

}

}
else
{
	if(input[i]>(105+offset1))
{
	if(input[i]==(107+offset1))
{
	vR[i] = 107;
}
else
{
	vR[i] = 106;
}

}
else
{
	if(input[i]==(105+offset1))
{
	vR[i] = 105;
}
else
{
	vR[i] = 104;
}

}

}

}
else
{
	if(input[i]>(99+offset1))
{
	if(input[i]>(101+offset1))
{
	if(input[i]==(103+offset1))
{
	vR[i] = 103;
}
else
{
	vR[i] = 102;
}

}
else
{
	if(input[i]==(101+offset1))
{
	vR[i] = 101;
}
else
{
	vR[i] = 100;
}

}

}
else
{
	if(input[i]>(97+offset1))
{
	if(input[i]==(99+offset1))
{
	vR[i] = 99;
}
else
{
	vR[i] = 98;
}

}
else
{
	if(input[i]==(97+offset1))
{
	vR[i] = 97;
}
else
{
	vR[i] = 96;
}

}

}

}

}

}
else
{
	if(input[i]>(79+offset1))
{
	if(input[i]>(87+offset1))
{
	if(input[i]>(91+offset1))
{
	if(input[i]>(93+offset1))
{
	if(input[i]==(95+offset1))
{
	vR[i] = 95;
}
else
{
	vR[i] = 94;
}

}
else
{
	if(input[i]==(93+offset1))
{
	vR[i] = 93;
}
else
{
	vR[i] = 92;
}

}

}
else
{
	if(input[i]>(89+offset1))
{
	if(input[i]==(91+offset1))
{
	vR[i] = 91;
}
else
{
	vR[i] = 90;
}

}
else
{
	if(input[i]==(89+offset1))
{
	vR[i] = 89;
}
else
{
	vR[i] = 88;
}

}

}

}
else
{
	if(input[i]>(83+offset1))
{
	if(input[i]>(85+offset1))
{
	if(input[i]==(87+offset1))
{
	vR[i] = 87;
}
else
{
	vR[i] = 86;
}

}
else
{
	if(input[i]==(85+offset1))
{
	vR[i] = 85;
}
else
{
	vR[i] = 84;
}

}

}
else
{
	if(input[i]>(81+offset1))
{
	if(input[i]==(83+offset1))
{
	vR[i] = 83;
}
else
{
	vR[i] = 82;
}

}
else
{
	if(input[i]==(81+offset1))
{
	vR[i] = 81;
}
else
{
	vR[i] = 80;
}

}

}

}

}
else
{
	if(input[i]>(71+offset1))
{
	if(input[i]>(75+offset1))
{
	if(input[i]>(77+offset1))
{
	if(input[i]==(79+offset1))
{
	vR[i] = 79;
}
else
{
	vR[i] = 78;
}

}
else
{
	if(input[i]==(77+offset1))
{
	vR[i] = 77;
}
else
{
	vR[i] = 76;
}

}

}
else
{
	if(input[i]>(73+offset1))
{
	if(input[i]==(75+offset1))
{
	vR[i] = 75;
}
else
{
	vR[i] = 74;
}

}
else
{
	if(input[i]==(73+offset1))
{
	vR[i] = 73;
}
else
{
	vR[i] = 72;
}

}

}

}
else
{
	if(input[i]>(67+offset1))
{
	if(input[i]>(69+offset1))
{
	if(input[i]==(71+offset1))
{
	vR[i] = 71;
}
else
{
	vR[i] = 70;
}

}
else
{
	if(input[i]==(69+offset1))
{
	vR[i] = 69;
}
else
{
	vR[i] = 68;
}

}

}
else
{
	if(input[i]>(65+offset1))
{
	if(input[i]==(67+offset1))
{
	vR[i] = 67;
}
else
{
	vR[i] = 66;
}

}
else
{
	if(input[i]==(65+offset1))
{
	vR[i] = 65;
}
else
{
	vR[i] = 64;
}

}

}

}

}

}

}
else
{
	if(input[i]>(31+offset1))
{
	if(input[i]>(47+offset1))
{
	if(input[i]>(55+offset1))
{
	if(input[i]>(59+offset1))
{
	if(input[i]>(61+offset1))
{
	if(input[i]==(63+offset1))
{
	vR[i] = 63;
}
else
{
	vR[i] = 62;
}

}
else
{
	if(input[i]==(61+offset1))
{
	vR[i] = 61;
}
else
{
	vR[i] = 60;
}

}

}
else
{
	if(input[i]>(57+offset1))
{
	if(input[i]==(59+offset1))
{
	vR[i] = 59;
}
else
{
	vR[i] = 58;
}

}
else
{
	if(input[i]==(57+offset1))
{
	vR[i] = 57;
}
else
{
	vR[i] = 56;
}

}

}

}
else
{
	if(input[i]>(51+offset1))
{
	if(input[i]>(53+offset1))
{
	if(input[i]==(55+offset1))
{
	vR[i] = 55;
}
else
{
	vR[i] = 54;
}

}
else
{
	if(input[i]==(53+offset1))
{
	vR[i] = 53;
}
else
{
	vR[i] = 52;
}

}

}
else
{
	if(input[i]>(49+offset1))
{
	if(input[i]==(51+offset1))
{
	vR[i] = 51;
}
else
{
	vR[i] = 50;
}

}
else
{
	if(input[i]==(49+offset1))
{
	vR[i] = 49;
}
else
{
	vR[i] = 48;
}

}

}

}

}
else
{
	if(input[i]>(39+offset1))
{
	if(input[i]>(43+offset1))
{
	if(input[i]>(45+offset1))
{
	if(input[i]==(47+offset1))
{
	vR[i] = 47;
}
else
{
	vR[i] = 46;
}

}
else
{
	if(input[i]==(45+offset1))
{
	vR[i] = 45;
}
else
{
	vR[i] = 44;
}

}

}
else
{
	if(input[i]>(41+offset1))
{
	if(input[i]==(43+offset1))
{
	vR[i] = 43;
}
else
{
	vR[i] = 42;
}

}
else
{
	if(input[i]==(41+offset1))
{
	vR[i] = 41;
}
else
{
	vR[i] = 40;
}

}

}

}
else
{
	if(input[i]>(35+offset1))
{
	if(input[i]>(37+offset1))
{
	if(input[i]==(39+offset1))
{
	vR[i] = 39;
}
else
{
	vR[i] = 38;
}

}
else
{
	if(input[i]==(37+offset1))
{
	vR[i] = 37;
}
else
{
	vR[i] = 36;
}

}

}
else
{
	if(input[i]>(33+offset1))
{
	if(input[i]==(35+offset1))
{
	vR[i] = 35;
}
else
{
	vR[i] = 34;
}

}
else
{
	if(input[i]==(33+offset1))
{
	vR[i] = 33;
}
else
{
	vR[i] = 32;
}

}

}

}

}

}
else
{
	if(input[i]>(15+offset1))
{
	if(input[i]>(23+offset1))
{
	if(input[i]>(27+offset1))
{
	if(input[i]>(29+offset1))
{
	if(input[i]==(31+offset1))
{
	vR[i] = 31;
}
else
{
	vR[i] = 30;
}

}
else
{
	if(input[i]==(29+offset1))
{
	vR[i] = 29;
}
else
{
	vR[i] = 28;
}

}

}
else
{
	if(input[i]>(25+offset1))
{
	if(input[i]==(27+offset1))
{
	vR[i] = 27;
}
else
{
	vR[i] = 26;
}

}
else
{
	if(input[i]==(25+offset1))
{
	vR[i] = 25;
}
else
{
	vR[i] = 24;
}

}

}

}
else
{
	if(input[i]>(19+offset1))
{
	if(input[i]>(21+offset1))
{
	if(input[i]==(23+offset1))
{
	vR[i] = 23;
}
else
{
	vR[i] = 22;
}

}
else
{
	if(input[i]==(21+offset1))
{
	vR[i] = 21;
}
else
{
	vR[i] = 20;
}

}

}
else
{
	if(input[i]>(17+offset1))
{
	if(input[i]==(19+offset1))
{
	vR[i] = 19;
}
else
{
	vR[i] = 18;
}

}
else
{
	if(input[i]==(17+offset1))
{
	vR[i] = 17;
}
else
{
	vR[i] = 16;
}

}

}

}

}
else
{
	if(input[i]>(7+offset1))
{
	if(input[i]>(11+offset1))
{
	if(input[i]>(13+offset1))
{
	if(input[i]==(15+offset1))
{
	vR[i] = 15;
}
else
{
	vR[i] = 14;
}

}
else
{
	if(input[i]==(13+offset1))
{
	vR[i] = 13;
}
else
{
	vR[i] = 12;
}

}

}
else
{
	if(input[i]>(9+offset1))
{
	if(input[i]==(11+offset1))
{
	vR[i] = 11;
}
else
{
	vR[i] = 10;
}

}
else
{
	if(input[i]==(9+offset1))
{
	vR[i] = 9;
}
else
{
	vR[i] = 8;
}

}

}

}
else
{
	if(input[i]>(3+offset1))
{
	if(input[i]>(5+offset1))
{
	if(input[i]==(7+offset1))
{
	vR[i] = 7;
}
else
{
	vR[i] = 6;
}

}
else
{
	if(input[i]==(5+offset1))
{
	vR[i] = 5;
}
else
{
	vR[i] = 4;
}

}

}
else
{
	if(input[i]>(1+offset1))
{
	if(input[i]==(3+offset1))
{
	vR[i] = 3;
}
else
{
	vR[i] = 2;
}

}
else
{
	if(input[i]==(1+offset1))
{
	vR[i] = 1;
}
else
{
	vR[i] = 0;
}

}

}

}

}

}

}

}

}

}

}


if(input[i]>(1023+offset2))
{
	if(input[i]>(1535+offset2))
{
	if(input[i]>(1791+offset2))
{
	if(input[i]>(1919+offset2))
{
	if(input[i]>(1983+offset2))
{
	if(input[i]>(2015+offset2))
{
	if(input[i]>(2031+offset2))
{
	if(input[i]>(2039+offset2))
{
	if(input[i]>(2043+offset2))
{
	if(input[i]>(2045+offset2))
{
	if(input[i]==(2047+offset2))
{
	vR[i] = 2047;
}
else
{
	vR[i] = 2046;
}

}
else
{
	if(input[i]==(2045+offset2))
{
	vR[i] = 2045;
}
else
{
	vR[i] = 2044;
}

}

}
else
{
	if(input[i]>(2041+offset2))
{
	if(input[i]==(2043+offset2))
{
	vR[i] = 2043;
}
else
{
	vR[i] = 2042;
}

}
else
{
	if(input[i]==(2041+offset2))
{
	vR[i] = 2041;
}
else
{
	vR[i] = 2040;
}

}

}

}
else
{
	if(input[i]>(2035+offset2))
{
	if(input[i]>(2037+offset2))
{
	if(input[i]==(2039+offset2))
{
	vR[i] = 2039;
}
else
{
	vR[i] = 2038;
}

}
else
{
	if(input[i]==(2037+offset2))
{
	vR[i] = 2037;
}
else
{
	vR[i] = 2036;
}

}

}
else
{
	if(input[i]>(2033+offset2))
{
	if(input[i]==(2035+offset2))
{
	vR[i] = 2035;
}
else
{
	vR[i] = 2034;
}

}
else
{
	if(input[i]==(2033+offset2))
{
	vR[i] = 2033;
}
else
{
	vR[i] = 2032;
}

}

}

}

}
else
{
	if(input[i]>(2023+offset2))
{
	if(input[i]>(2027+offset2))
{
	if(input[i]>(2029+offset2))
{
	if(input[i]==(2031+offset2))
{
	vR[i] = 2031;
}
else
{
	vR[i] = 2030;
}

}
else
{
	if(input[i]==(2029+offset2))
{
	vR[i] = 2029;
}
else
{
	vR[i] = 2028;
}

}

}
else
{
	if(input[i]>(2025+offset2))
{
	if(input[i]==(2027+offset2))
{
	vR[i] = 2027;
}
else
{
	vR[i] = 2026;
}

}
else
{
	if(input[i]==(2025+offset2))
{
	vR[i] = 2025;
}
else
{
	vR[i] = 2024;
}

}

}

}
else
{
	if(input[i]>(2019+offset2))
{
	if(input[i]>(2021+offset2))
{
	if(input[i]==(2023+offset2))
{
	vR[i] = 2023;
}
else
{
	vR[i] = 2022;
}

}
else
{
	if(input[i]==(2021+offset2))
{
	vR[i] = 2021;
}
else
{
	vR[i] = 2020;
}

}

}
else
{
	if(input[i]>(2017+offset2))
{
	if(input[i]==(2019+offset2))
{
	vR[i] = 2019;
}
else
{
	vR[i] = 2018;
}

}
else
{
	if(input[i]==(2017+offset2))
{
	vR[i] = 2017;
}
else
{
	vR[i] = 2016;
}

}

}

}

}

}
else
{
	if(input[i]>(1999+offset2))
{
	if(input[i]>(2007+offset2))
{
	if(input[i]>(2011+offset2))
{
	if(input[i]>(2013+offset2))
{
	if(input[i]==(2015+offset2))
{
	vR[i] = 2015;
}
else
{
	vR[i] = 2014;
}

}
else
{
	if(input[i]==(2013+offset2))
{
	vR[i] = 2013;
}
else
{
	vR[i] = 2012;
}

}

}
else
{
	if(input[i]>(2009+offset2))
{
	if(input[i]==(2011+offset2))
{
	vR[i] = 2011;
}
else
{
	vR[i] = 2010;
}

}
else
{
	if(input[i]==(2009+offset2))
{
	vR[i] = 2009;
}
else
{
	vR[i] = 2008;
}

}

}

}
else
{
	if(input[i]>(2003+offset2))
{
	if(input[i]>(2005+offset2))
{
	if(input[i]==(2007+offset2))
{
	vR[i] = 2007;
}
else
{
	vR[i] = 2006;
}

}
else
{
	if(input[i]==(2005+offset2))
{
	vR[i] = 2005;
}
else
{
	vR[i] = 2004;
}

}

}
else
{
	if(input[i]>(2001+offset2))
{
	if(input[i]==(2003+offset2))
{
	vR[i] = 2003;
}
else
{
	vR[i] = 2002;
}

}
else
{
	if(input[i]==(2001+offset2))
{
	vR[i] = 2001;
}
else
{
	vR[i] = 2000;
}

}

}

}

}
else
{
	if(input[i]>(1991+offset2))
{
	if(input[i]>(1995+offset2))
{
	if(input[i]>(1997+offset2))
{
	if(input[i]==(1999+offset2))
{
	vR[i] = 1999;
}
else
{
	vR[i] = 1998;
}

}
else
{
	if(input[i]==(1997+offset2))
{
	vR[i] = 1997;
}
else
{
	vR[i] = 1996;
}

}

}
else
{
	if(input[i]>(1993+offset2))
{
	if(input[i]==(1995+offset2))
{
	vR[i] = 1995;
}
else
{
	vR[i] = 1994;
}

}
else
{
	if(input[i]==(1993+offset2))
{
	vR[i] = 1993;
}
else
{
	vR[i] = 1992;
}

}

}

}
else
{
	if(input[i]>(1987+offset2))
{
	if(input[i]>(1989+offset2))
{
	if(input[i]==(1991+offset2))
{
	vR[i] = 1991;
}
else
{
	vR[i] = 1990;
}

}
else
{
	if(input[i]==(1989+offset2))
{
	vR[i] = 1989;
}
else
{
	vR[i] = 1988;
}

}

}
else
{
	if(input[i]>(1985+offset2))
{
	if(input[i]==(1987+offset2))
{
	vR[i] = 1987;
}
else
{
	vR[i] = 1986;
}

}
else
{
	if(input[i]==(1985+offset2))
{
	vR[i] = 1985;
}
else
{
	vR[i] = 1984;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1951+offset2))
{
	if(input[i]>(1967+offset2))
{
	if(input[i]>(1975+offset2))
{
	if(input[i]>(1979+offset2))
{
	if(input[i]>(1981+offset2))
{
	if(input[i]==(1983+offset2))
{
	vR[i] = 1983;
}
else
{
	vR[i] = 1982;
}

}
else
{
	if(input[i]==(1981+offset2))
{
	vR[i] = 1981;
}
else
{
	vR[i] = 1980;
}

}

}
else
{
	if(input[i]>(1977+offset2))
{
	if(input[i]==(1979+offset2))
{
	vR[i] = 1979;
}
else
{
	vR[i] = 1978;
}

}
else
{
	if(input[i]==(1977+offset2))
{
	vR[i] = 1977;
}
else
{
	vR[i] = 1976;
}

}

}

}
else
{
	if(input[i]>(1971+offset2))
{
	if(input[i]>(1973+offset2))
{
	if(input[i]==(1975+offset2))
{
	vR[i] = 1975;
}
else
{
	vR[i] = 1974;
}

}
else
{
	if(input[i]==(1973+offset2))
{
	vR[i] = 1973;
}
else
{
	vR[i] = 1972;
}

}

}
else
{
	if(input[i]>(1969+offset2))
{
	if(input[i]==(1971+offset2))
{
	vR[i] = 1971;
}
else
{
	vR[i] = 1970;
}

}
else
{
	if(input[i]==(1969+offset2))
{
	vR[i] = 1969;
}
else
{
	vR[i] = 1968;
}

}

}

}

}
else
{
	if(input[i]>(1959+offset2))
{
	if(input[i]>(1963+offset2))
{
	if(input[i]>(1965+offset2))
{
	if(input[i]==(1967+offset2))
{
	vR[i] = 1967;
}
else
{
	vR[i] = 1966;
}

}
else
{
	if(input[i]==(1965+offset2))
{
	vR[i] = 1965;
}
else
{
	vR[i] = 1964;
}

}

}
else
{
	if(input[i]>(1961+offset2))
{
	if(input[i]==(1963+offset2))
{
	vR[i] = 1963;
}
else
{
	vR[i] = 1962;
}

}
else
{
	if(input[i]==(1961+offset2))
{
	vR[i] = 1961;
}
else
{
	vR[i] = 1960;
}

}

}

}
else
{
	if(input[i]>(1955+offset2))
{
	if(input[i]>(1957+offset2))
{
	if(input[i]==(1959+offset2))
{
	vR[i] = 1959;
}
else
{
	vR[i] = 1958;
}

}
else
{
	if(input[i]==(1957+offset2))
{
	vR[i] = 1957;
}
else
{
	vR[i] = 1956;
}

}

}
else
{
	if(input[i]>(1953+offset2))
{
	if(input[i]==(1955+offset2))
{
	vR[i] = 1955;
}
else
{
	vR[i] = 1954;
}

}
else
{
	if(input[i]==(1953+offset2))
{
	vR[i] = 1953;
}
else
{
	vR[i] = 1952;
}

}

}

}

}

}
else
{
	if(input[i]>(1935+offset2))
{
	if(input[i]>(1943+offset2))
{
	if(input[i]>(1947+offset2))
{
	if(input[i]>(1949+offset2))
{
	if(input[i]==(1951+offset2))
{
	vR[i] = 1951;
}
else
{
	vR[i] = 1950;
}

}
else
{
	if(input[i]==(1949+offset2))
{
	vR[i] = 1949;
}
else
{
	vR[i] = 1948;
}

}

}
else
{
	if(input[i]>(1945+offset2))
{
	if(input[i]==(1947+offset2))
{
	vR[i] = 1947;
}
else
{
	vR[i] = 1946;
}

}
else
{
	if(input[i]==(1945+offset2))
{
	vR[i] = 1945;
}
else
{
	vR[i] = 1944;
}

}

}

}
else
{
	if(input[i]>(1939+offset2))
{
	if(input[i]>(1941+offset2))
{
	if(input[i]==(1943+offset2))
{
	vR[i] = 1943;
}
else
{
	vR[i] = 1942;
}

}
else
{
	if(input[i]==(1941+offset2))
{
	vR[i] = 1941;
}
else
{
	vR[i] = 1940;
}

}

}
else
{
	if(input[i]>(1937+offset2))
{
	if(input[i]==(1939+offset2))
{
	vR[i] = 1939;
}
else
{
	vR[i] = 1938;
}

}
else
{
	if(input[i]==(1937+offset2))
{
	vR[i] = 1937;
}
else
{
	vR[i] = 1936;
}

}

}

}

}
else
{
	if(input[i]>(1927+offset2))
{
	if(input[i]>(1931+offset2))
{
	if(input[i]>(1933+offset2))
{
	if(input[i]==(1935+offset2))
{
	vR[i] = 1935;
}
else
{
	vR[i] = 1934;
}

}
else
{
	if(input[i]==(1933+offset2))
{
	vR[i] = 1933;
}
else
{
	vR[i] = 1932;
}

}

}
else
{
	if(input[i]>(1929+offset2))
{
	if(input[i]==(1931+offset2))
{
	vR[i] = 1931;
}
else
{
	vR[i] = 1930;
}

}
else
{
	if(input[i]==(1929+offset2))
{
	vR[i] = 1929;
}
else
{
	vR[i] = 1928;
}

}

}

}
else
{
	if(input[i]>(1923+offset2))
{
	if(input[i]>(1925+offset2))
{
	if(input[i]==(1927+offset2))
{
	vR[i] = 1927;
}
else
{
	vR[i] = 1926;
}

}
else
{
	if(input[i]==(1925+offset2))
{
	vR[i] = 1925;
}
else
{
	vR[i] = 1924;
}

}

}
else
{
	if(input[i]>(1921+offset2))
{
	if(input[i]==(1923+offset2))
{
	vR[i] = 1923;
}
else
{
	vR[i] = 1922;
}

}
else
{
	if(input[i]==(1921+offset2))
{
	vR[i] = 1921;
}
else
{
	vR[i] = 1920;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(1855+offset2))
{
	if(input[i]>(1887+offset2))
{
	if(input[i]>(1903+offset2))
{
	if(input[i]>(1911+offset2))
{
	if(input[i]>(1915+offset2))
{
	if(input[i]>(1917+offset2))
{
	if(input[i]==(1919+offset2))
{
	vR[i] = 1919;
}
else
{
	vR[i] = 1918;
}

}
else
{
	if(input[i]==(1917+offset2))
{
	vR[i] = 1917;
}
else
{
	vR[i] = 1916;
}

}

}
else
{
	if(input[i]>(1913+offset2))
{
	if(input[i]==(1915+offset2))
{
	vR[i] = 1915;
}
else
{
	vR[i] = 1914;
}

}
else
{
	if(input[i]==(1913+offset2))
{
	vR[i] = 1913;
}
else
{
	vR[i] = 1912;
}

}

}

}
else
{
	if(input[i]>(1907+offset2))
{
	if(input[i]>(1909+offset2))
{
	if(input[i]==(1911+offset2))
{
	vR[i] = 1911;
}
else
{
	vR[i] = 1910;
}

}
else
{
	if(input[i]==(1909+offset2))
{
	vR[i] = 1909;
}
else
{
	vR[i] = 1908;
}

}

}
else
{
	if(input[i]>(1905+offset2))
{
	if(input[i]==(1907+offset2))
{
	vR[i] = 1907;
}
else
{
	vR[i] = 1906;
}

}
else
{
	if(input[i]==(1905+offset2))
{
	vR[i] = 1905;
}
else
{
	vR[i] = 1904;
}

}

}

}

}
else
{
	if(input[i]>(1895+offset2))
{
	if(input[i]>(1899+offset2))
{
	if(input[i]>(1901+offset2))
{
	if(input[i]==(1903+offset2))
{
	vR[i] = 1903;
}
else
{
	vR[i] = 1902;
}

}
else
{
	if(input[i]==(1901+offset2))
{
	vR[i] = 1901;
}
else
{
	vR[i] = 1900;
}

}

}
else
{
	if(input[i]>(1897+offset2))
{
	if(input[i]==(1899+offset2))
{
	vR[i] = 1899;
}
else
{
	vR[i] = 1898;
}

}
else
{
	if(input[i]==(1897+offset2))
{
	vR[i] = 1897;
}
else
{
	vR[i] = 1896;
}

}

}

}
else
{
	if(input[i]>(1891+offset2))
{
	if(input[i]>(1893+offset2))
{
	if(input[i]==(1895+offset2))
{
	vR[i] = 1895;
}
else
{
	vR[i] = 1894;
}

}
else
{
	if(input[i]==(1893+offset2))
{
	vR[i] = 1893;
}
else
{
	vR[i] = 1892;
}

}

}
else
{
	if(input[i]>(1889+offset2))
{
	if(input[i]==(1891+offset2))
{
	vR[i] = 1891;
}
else
{
	vR[i] = 1890;
}

}
else
{
	if(input[i]==(1889+offset2))
{
	vR[i] = 1889;
}
else
{
	vR[i] = 1888;
}

}

}

}

}

}
else
{
	if(input[i]>(1871+offset2))
{
	if(input[i]>(1879+offset2))
{
	if(input[i]>(1883+offset2))
{
	if(input[i]>(1885+offset2))
{
	if(input[i]==(1887+offset2))
{
	vR[i] = 1887;
}
else
{
	vR[i] = 1886;
}

}
else
{
	if(input[i]==(1885+offset2))
{
	vR[i] = 1885;
}
else
{
	vR[i] = 1884;
}

}

}
else
{
	if(input[i]>(1881+offset2))
{
	if(input[i]==(1883+offset2))
{
	vR[i] = 1883;
}
else
{
	vR[i] = 1882;
}

}
else
{
	if(input[i]==(1881+offset2))
{
	vR[i] = 1881;
}
else
{
	vR[i] = 1880;
}

}

}

}
else
{
	if(input[i]>(1875+offset2))
{
	if(input[i]>(1877+offset2))
{
	if(input[i]==(1879+offset2))
{
	vR[i] = 1879;
}
else
{
	vR[i] = 1878;
}

}
else
{
	if(input[i]==(1877+offset2))
{
	vR[i] = 1877;
}
else
{
	vR[i] = 1876;
}

}

}
else
{
	if(input[i]>(1873+offset2))
{
	if(input[i]==(1875+offset2))
{
	vR[i] = 1875;
}
else
{
	vR[i] = 1874;
}

}
else
{
	if(input[i]==(1873+offset2))
{
	vR[i] = 1873;
}
else
{
	vR[i] = 1872;
}

}

}

}

}
else
{
	if(input[i]>(1863+offset2))
{
	if(input[i]>(1867+offset2))
{
	if(input[i]>(1869+offset2))
{
	if(input[i]==(1871+offset2))
{
	vR[i] = 1871;
}
else
{
	vR[i] = 1870;
}

}
else
{
	if(input[i]==(1869+offset2))
{
	vR[i] = 1869;
}
else
{
	vR[i] = 1868;
}

}

}
else
{
	if(input[i]>(1865+offset2))
{
	if(input[i]==(1867+offset2))
{
	vR[i] = 1867;
}
else
{
	vR[i] = 1866;
}

}
else
{
	if(input[i]==(1865+offset2))
{
	vR[i] = 1865;
}
else
{
	vR[i] = 1864;
}

}

}

}
else
{
	if(input[i]>(1859+offset2))
{
	if(input[i]>(1861+offset2))
{
	if(input[i]==(1863+offset2))
{
	vR[i] = 1863;
}
else
{
	vR[i] = 1862;
}

}
else
{
	if(input[i]==(1861+offset2))
{
	vR[i] = 1861;
}
else
{
	vR[i] = 1860;
}

}

}
else
{
	if(input[i]>(1857+offset2))
{
	if(input[i]==(1859+offset2))
{
	vR[i] = 1859;
}
else
{
	vR[i] = 1858;
}

}
else
{
	if(input[i]==(1857+offset2))
{
	vR[i] = 1857;
}
else
{
	vR[i] = 1856;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1823+offset2))
{
	if(input[i]>(1839+offset2))
{
	if(input[i]>(1847+offset2))
{
	if(input[i]>(1851+offset2))
{
	if(input[i]>(1853+offset2))
{
	if(input[i]==(1855+offset2))
{
	vR[i] = 1855;
}
else
{
	vR[i] = 1854;
}

}
else
{
	if(input[i]==(1853+offset2))
{
	vR[i] = 1853;
}
else
{
	vR[i] = 1852;
}

}

}
else
{
	if(input[i]>(1849+offset2))
{
	if(input[i]==(1851+offset2))
{
	vR[i] = 1851;
}
else
{
	vR[i] = 1850;
}

}
else
{
	if(input[i]==(1849+offset2))
{
	vR[i] = 1849;
}
else
{
	vR[i] = 1848;
}

}

}

}
else
{
	if(input[i]>(1843+offset2))
{
	if(input[i]>(1845+offset2))
{
	if(input[i]==(1847+offset2))
{
	vR[i] = 1847;
}
else
{
	vR[i] = 1846;
}

}
else
{
	if(input[i]==(1845+offset2))
{
	vR[i] = 1845;
}
else
{
	vR[i] = 1844;
}

}

}
else
{
	if(input[i]>(1841+offset2))
{
	if(input[i]==(1843+offset2))
{
	vR[i] = 1843;
}
else
{
	vR[i] = 1842;
}

}
else
{
	if(input[i]==(1841+offset2))
{
	vR[i] = 1841;
}
else
{
	vR[i] = 1840;
}

}

}

}

}
else
{
	if(input[i]>(1831+offset2))
{
	if(input[i]>(1835+offset2))
{
	if(input[i]>(1837+offset2))
{
	if(input[i]==(1839+offset2))
{
	vR[i] = 1839;
}
else
{
	vR[i] = 1838;
}

}
else
{
	if(input[i]==(1837+offset2))
{
	vR[i] = 1837;
}
else
{
	vR[i] = 1836;
}

}

}
else
{
	if(input[i]>(1833+offset2))
{
	if(input[i]==(1835+offset2))
{
	vR[i] = 1835;
}
else
{
	vR[i] = 1834;
}

}
else
{
	if(input[i]==(1833+offset2))
{
	vR[i] = 1833;
}
else
{
	vR[i] = 1832;
}

}

}

}
else
{
	if(input[i]>(1827+offset2))
{
	if(input[i]>(1829+offset2))
{
	if(input[i]==(1831+offset2))
{
	vR[i] = 1831;
}
else
{
	vR[i] = 1830;
}

}
else
{
	if(input[i]==(1829+offset2))
{
	vR[i] = 1829;
}
else
{
	vR[i] = 1828;
}

}

}
else
{
	if(input[i]>(1825+offset2))
{
	if(input[i]==(1827+offset2))
{
	vR[i] = 1827;
}
else
{
	vR[i] = 1826;
}

}
else
{
	if(input[i]==(1825+offset2))
{
	vR[i] = 1825;
}
else
{
	vR[i] = 1824;
}

}

}

}

}

}
else
{
	if(input[i]>(1807+offset2))
{
	if(input[i]>(1815+offset2))
{
	if(input[i]>(1819+offset2))
{
	if(input[i]>(1821+offset2))
{
	if(input[i]==(1823+offset2))
{
	vR[i] = 1823;
}
else
{
	vR[i] = 1822;
}

}
else
{
	if(input[i]==(1821+offset2))
{
	vR[i] = 1821;
}
else
{
	vR[i] = 1820;
}

}

}
else
{
	if(input[i]>(1817+offset2))
{
	if(input[i]==(1819+offset2))
{
	vR[i] = 1819;
}
else
{
	vR[i] = 1818;
}

}
else
{
	if(input[i]==(1817+offset2))
{
	vR[i] = 1817;
}
else
{
	vR[i] = 1816;
}

}

}

}
else
{
	if(input[i]>(1811+offset2))
{
	if(input[i]>(1813+offset2))
{
	if(input[i]==(1815+offset2))
{
	vR[i] = 1815;
}
else
{
	vR[i] = 1814;
}

}
else
{
	if(input[i]==(1813+offset2))
{
	vR[i] = 1813;
}
else
{
	vR[i] = 1812;
}

}

}
else
{
	if(input[i]>(1809+offset2))
{
	if(input[i]==(1811+offset2))
{
	vR[i] = 1811;
}
else
{
	vR[i] = 1810;
}

}
else
{
	if(input[i]==(1809+offset2))
{
	vR[i] = 1809;
}
else
{
	vR[i] = 1808;
}

}

}

}

}
else
{
	if(input[i]>(1799+offset2))
{
	if(input[i]>(1803+offset2))
{
	if(input[i]>(1805+offset2))
{
	if(input[i]==(1807+offset2))
{
	vR[i] = 1807;
}
else
{
	vR[i] = 1806;
}

}
else
{
	if(input[i]==(1805+offset2))
{
	vR[i] = 1805;
}
else
{
	vR[i] = 1804;
}

}

}
else
{
	if(input[i]>(1801+offset2))
{
	if(input[i]==(1803+offset2))
{
	vR[i] = 1803;
}
else
{
	vR[i] = 1802;
}

}
else
{
	if(input[i]==(1801+offset2))
{
	vR[i] = 1801;
}
else
{
	vR[i] = 1800;
}

}

}

}
else
{
	if(input[i]>(1795+offset2))
{
	if(input[i]>(1797+offset2))
{
	if(input[i]==(1799+offset2))
{
	vR[i] = 1799;
}
else
{
	vR[i] = 1798;
}

}
else
{
	if(input[i]==(1797+offset2))
{
	vR[i] = 1797;
}
else
{
	vR[i] = 1796;
}

}

}
else
{
	if(input[i]>(1793+offset2))
{
	if(input[i]==(1795+offset2))
{
	vR[i] = 1795;
}
else
{
	vR[i] = 1794;
}

}
else
{
	if(input[i]==(1793+offset2))
{
	vR[i] = 1793;
}
else
{
	vR[i] = 1792;
}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(1663+offset2))
{
	if(input[i]>(1727+offset2))
{
	if(input[i]>(1759+offset2))
{
	if(input[i]>(1775+offset2))
{
	if(input[i]>(1783+offset2))
{
	if(input[i]>(1787+offset2))
{
	if(input[i]>(1789+offset2))
{
	if(input[i]==(1791+offset2))
{
	vR[i] = 1791;
}
else
{
	vR[i] = 1790;
}

}
else
{
	if(input[i]==(1789+offset2))
{
	vR[i] = 1789;
}
else
{
	vR[i] = 1788;
}

}

}
else
{
	if(input[i]>(1785+offset2))
{
	if(input[i]==(1787+offset2))
{
	vR[i] = 1787;
}
else
{
	vR[i] = 1786;
}

}
else
{
	if(input[i]==(1785+offset2))
{
	vR[i] = 1785;
}
else
{
	vR[i] = 1784;
}

}

}

}
else
{
	if(input[i]>(1779+offset2))
{
	if(input[i]>(1781+offset2))
{
	if(input[i]==(1783+offset2))
{
	vR[i] = 1783;
}
else
{
	vR[i] = 1782;
}

}
else
{
	if(input[i]==(1781+offset2))
{
	vR[i] = 1781;
}
else
{
	vR[i] = 1780;
}

}

}
else
{
	if(input[i]>(1777+offset2))
{
	if(input[i]==(1779+offset2))
{
	vR[i] = 1779;
}
else
{
	vR[i] = 1778;
}

}
else
{
	if(input[i]==(1777+offset2))
{
	vR[i] = 1777;
}
else
{
	vR[i] = 1776;
}

}

}

}

}
else
{
	if(input[i]>(1767+offset2))
{
	if(input[i]>(1771+offset2))
{
	if(input[i]>(1773+offset2))
{
	if(input[i]==(1775+offset2))
{
	vR[i] = 1775;
}
else
{
	vR[i] = 1774;
}

}
else
{
	if(input[i]==(1773+offset2))
{
	vR[i] = 1773;
}
else
{
	vR[i] = 1772;
}

}

}
else
{
	if(input[i]>(1769+offset2))
{
	if(input[i]==(1771+offset2))
{
	vR[i] = 1771;
}
else
{
	vR[i] = 1770;
}

}
else
{
	if(input[i]==(1769+offset2))
{
	vR[i] = 1769;
}
else
{
	vR[i] = 1768;
}

}

}

}
else
{
	if(input[i]>(1763+offset2))
{
	if(input[i]>(1765+offset2))
{
	if(input[i]==(1767+offset2))
{
	vR[i] = 1767;
}
else
{
	vR[i] = 1766;
}

}
else
{
	if(input[i]==(1765+offset2))
{
	vR[i] = 1765;
}
else
{
	vR[i] = 1764;
}

}

}
else
{
	if(input[i]>(1761+offset2))
{
	if(input[i]==(1763+offset2))
{
	vR[i] = 1763;
}
else
{
	vR[i] = 1762;
}

}
else
{
	if(input[i]==(1761+offset2))
{
	vR[i] = 1761;
}
else
{
	vR[i] = 1760;
}

}

}

}

}

}
else
{
	if(input[i]>(1743+offset2))
{
	if(input[i]>(1751+offset2))
{
	if(input[i]>(1755+offset2))
{
	if(input[i]>(1757+offset2))
{
	if(input[i]==(1759+offset2))
{
	vR[i] = 1759;
}
else
{
	vR[i] = 1758;
}

}
else
{
	if(input[i]==(1757+offset2))
{
	vR[i] = 1757;
}
else
{
	vR[i] = 1756;
}

}

}
else
{
	if(input[i]>(1753+offset2))
{
	if(input[i]==(1755+offset2))
{
	vR[i] = 1755;
}
else
{
	vR[i] = 1754;
}

}
else
{
	if(input[i]==(1753+offset2))
{
	vR[i] = 1753;
}
else
{
	vR[i] = 1752;
}

}

}

}
else
{
	if(input[i]>(1747+offset2))
{
	if(input[i]>(1749+offset2))
{
	if(input[i]==(1751+offset2))
{
	vR[i] = 1751;
}
else
{
	vR[i] = 1750;
}

}
else
{
	if(input[i]==(1749+offset2))
{
	vR[i] = 1749;
}
else
{
	vR[i] = 1748;
}

}

}
else
{
	if(input[i]>(1745+offset2))
{
	if(input[i]==(1747+offset2))
{
	vR[i] = 1747;
}
else
{
	vR[i] = 1746;
}

}
else
{
	if(input[i]==(1745+offset2))
{
	vR[i] = 1745;
}
else
{
	vR[i] = 1744;
}

}

}

}

}
else
{
	if(input[i]>(1735+offset2))
{
	if(input[i]>(1739+offset2))
{
	if(input[i]>(1741+offset2))
{
	if(input[i]==(1743+offset2))
{
	vR[i] = 1743;
}
else
{
	vR[i] = 1742;
}

}
else
{
	if(input[i]==(1741+offset2))
{
	vR[i] = 1741;
}
else
{
	vR[i] = 1740;
}

}

}
else
{
	if(input[i]>(1737+offset2))
{
	if(input[i]==(1739+offset2))
{
	vR[i] = 1739;
}
else
{
	vR[i] = 1738;
}

}
else
{
	if(input[i]==(1737+offset2))
{
	vR[i] = 1737;
}
else
{
	vR[i] = 1736;
}

}

}

}
else
{
	if(input[i]>(1731+offset2))
{
	if(input[i]>(1733+offset2))
{
	if(input[i]==(1735+offset2))
{
	vR[i] = 1735;
}
else
{
	vR[i] = 1734;
}

}
else
{
	if(input[i]==(1733+offset2))
{
	vR[i] = 1733;
}
else
{
	vR[i] = 1732;
}

}

}
else
{
	if(input[i]>(1729+offset2))
{
	if(input[i]==(1731+offset2))
{
	vR[i] = 1731;
}
else
{
	vR[i] = 1730;
}

}
else
{
	if(input[i]==(1729+offset2))
{
	vR[i] = 1729;
}
else
{
	vR[i] = 1728;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1695+offset2))
{
	if(input[i]>(1711+offset2))
{
	if(input[i]>(1719+offset2))
{
	if(input[i]>(1723+offset2))
{
	if(input[i]>(1725+offset2))
{
	if(input[i]==(1727+offset2))
{
	vR[i] = 1727;
}
else
{
	vR[i] = 1726;
}

}
else
{
	if(input[i]==(1725+offset2))
{
	vR[i] = 1725;
}
else
{
	vR[i] = 1724;
}

}

}
else
{
	if(input[i]>(1721+offset2))
{
	if(input[i]==(1723+offset2))
{
	vR[i] = 1723;
}
else
{
	vR[i] = 1722;
}

}
else
{
	if(input[i]==(1721+offset2))
{
	vR[i] = 1721;
}
else
{
	vR[i] = 1720;
}

}

}

}
else
{
	if(input[i]>(1715+offset2))
{
	if(input[i]>(1717+offset2))
{
	if(input[i]==(1719+offset2))
{
	vR[i] = 1719;
}
else
{
	vR[i] = 1718;
}

}
else
{
	if(input[i]==(1717+offset2))
{
	vR[i] = 1717;
}
else
{
	vR[i] = 1716;
}

}

}
else
{
	if(input[i]>(1713+offset2))
{
	if(input[i]==(1715+offset2))
{
	vR[i] = 1715;
}
else
{
	vR[i] = 1714;
}

}
else
{
	if(input[i]==(1713+offset2))
{
	vR[i] = 1713;
}
else
{
	vR[i] = 1712;
}

}

}

}

}
else
{
	if(input[i]>(1703+offset2))
{
	if(input[i]>(1707+offset2))
{
	if(input[i]>(1709+offset2))
{
	if(input[i]==(1711+offset2))
{
	vR[i] = 1711;
}
else
{
	vR[i] = 1710;
}

}
else
{
	if(input[i]==(1709+offset2))
{
	vR[i] = 1709;
}
else
{
	vR[i] = 1708;
}

}

}
else
{
	if(input[i]>(1705+offset2))
{
	if(input[i]==(1707+offset2))
{
	vR[i] = 1707;
}
else
{
	vR[i] = 1706;
}

}
else
{
	if(input[i]==(1705+offset2))
{
	vR[i] = 1705;
}
else
{
	vR[i] = 1704;
}

}

}

}
else
{
	if(input[i]>(1699+offset2))
{
	if(input[i]>(1701+offset2))
{
	if(input[i]==(1703+offset2))
{
	vR[i] = 1703;
}
else
{
	vR[i] = 1702;
}

}
else
{
	if(input[i]==(1701+offset2))
{
	vR[i] = 1701;
}
else
{
	vR[i] = 1700;
}

}

}
else
{
	if(input[i]>(1697+offset2))
{
	if(input[i]==(1699+offset2))
{
	vR[i] = 1699;
}
else
{
	vR[i] = 1698;
}

}
else
{
	if(input[i]==(1697+offset2))
{
	vR[i] = 1697;
}
else
{
	vR[i] = 1696;
}

}

}

}

}

}
else
{
	if(input[i]>(1679+offset2))
{
	if(input[i]>(1687+offset2))
{
	if(input[i]>(1691+offset2))
{
	if(input[i]>(1693+offset2))
{
	if(input[i]==(1695+offset2))
{
	vR[i] = 1695;
}
else
{
	vR[i] = 1694;
}

}
else
{
	if(input[i]==(1693+offset2))
{
	vR[i] = 1693;
}
else
{
	vR[i] = 1692;
}

}

}
else
{
	if(input[i]>(1689+offset2))
{
	if(input[i]==(1691+offset2))
{
	vR[i] = 1691;
}
else
{
	vR[i] = 1690;
}

}
else
{
	if(input[i]==(1689+offset2))
{
	vR[i] = 1689;
}
else
{
	vR[i] = 1688;
}

}

}

}
else
{
	if(input[i]>(1683+offset2))
{
	if(input[i]>(1685+offset2))
{
	if(input[i]==(1687+offset2))
{
	vR[i] = 1687;
}
else
{
	vR[i] = 1686;
}

}
else
{
	if(input[i]==(1685+offset2))
{
	vR[i] = 1685;
}
else
{
	vR[i] = 1684;
}

}

}
else
{
	if(input[i]>(1681+offset2))
{
	if(input[i]==(1683+offset2))
{
	vR[i] = 1683;
}
else
{
	vR[i] = 1682;
}

}
else
{
	if(input[i]==(1681+offset2))
{
	vR[i] = 1681;
}
else
{
	vR[i] = 1680;
}

}

}

}

}
else
{
	if(input[i]>(1671+offset2))
{
	if(input[i]>(1675+offset2))
{
	if(input[i]>(1677+offset2))
{
	if(input[i]==(1679+offset2))
{
	vR[i] = 1679;
}
else
{
	vR[i] = 1678;
}

}
else
{
	if(input[i]==(1677+offset2))
{
	vR[i] = 1677;
}
else
{
	vR[i] = 1676;
}

}

}
else
{
	if(input[i]>(1673+offset2))
{
	if(input[i]==(1675+offset2))
{
	vR[i] = 1675;
}
else
{
	vR[i] = 1674;
}

}
else
{
	if(input[i]==(1673+offset2))
{
	vR[i] = 1673;
}
else
{
	vR[i] = 1672;
}

}

}

}
else
{
	if(input[i]>(1667+offset2))
{
	if(input[i]>(1669+offset2))
{
	if(input[i]==(1671+offset2))
{
	vR[i] = 1671;
}
else
{
	vR[i] = 1670;
}

}
else
{
	if(input[i]==(1669+offset2))
{
	vR[i] = 1669;
}
else
{
	vR[i] = 1668;
}

}

}
else
{
	if(input[i]>(1665+offset2))
{
	if(input[i]==(1667+offset2))
{
	vR[i] = 1667;
}
else
{
	vR[i] = 1666;
}

}
else
{
	if(input[i]==(1665+offset2))
{
	vR[i] = 1665;
}
else
{
	vR[i] = 1664;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(1599+offset2))
{
	if(input[i]>(1631+offset2))
{
	if(input[i]>(1647+offset2))
{
	if(input[i]>(1655+offset2))
{
	if(input[i]>(1659+offset2))
{
	if(input[i]>(1661+offset2))
{
	if(input[i]==(1663+offset2))
{
	vR[i] = 1663;
}
else
{
	vR[i] = 1662;
}

}
else
{
	if(input[i]==(1661+offset2))
{
	vR[i] = 1661;
}
else
{
	vR[i] = 1660;
}

}

}
else
{
	if(input[i]>(1657+offset2))
{
	if(input[i]==(1659+offset2))
{
	vR[i] = 1659;
}
else
{
	vR[i] = 1658;
}

}
else
{
	if(input[i]==(1657+offset2))
{
	vR[i] = 1657;
}
else
{
	vR[i] = 1656;
}

}

}

}
else
{
	if(input[i]>(1651+offset2))
{
	if(input[i]>(1653+offset2))
{
	if(input[i]==(1655+offset2))
{
	vR[i] = 1655;
}
else
{
	vR[i] = 1654;
}

}
else
{
	if(input[i]==(1653+offset2))
{
	vR[i] = 1653;
}
else
{
	vR[i] = 1652;
}

}

}
else
{
	if(input[i]>(1649+offset2))
{
	if(input[i]==(1651+offset2))
{
	vR[i] = 1651;
}
else
{
	vR[i] = 1650;
}

}
else
{
	if(input[i]==(1649+offset2))
{
	vR[i] = 1649;
}
else
{
	vR[i] = 1648;
}

}

}

}

}
else
{
	if(input[i]>(1639+offset2))
{
	if(input[i]>(1643+offset2))
{
	if(input[i]>(1645+offset2))
{
	if(input[i]==(1647+offset2))
{
	vR[i] = 1647;
}
else
{
	vR[i] = 1646;
}

}
else
{
	if(input[i]==(1645+offset2))
{
	vR[i] = 1645;
}
else
{
	vR[i] = 1644;
}

}

}
else
{
	if(input[i]>(1641+offset2))
{
	if(input[i]==(1643+offset2))
{
	vR[i] = 1643;
}
else
{
	vR[i] = 1642;
}

}
else
{
	if(input[i]==(1641+offset2))
{
	vR[i] = 1641;
}
else
{
	vR[i] = 1640;
}

}

}

}
else
{
	if(input[i]>(1635+offset2))
{
	if(input[i]>(1637+offset2))
{
	if(input[i]==(1639+offset2))
{
	vR[i] = 1639;
}
else
{
	vR[i] = 1638;
}

}
else
{
	if(input[i]==(1637+offset2))
{
	vR[i] = 1637;
}
else
{
	vR[i] = 1636;
}

}

}
else
{
	if(input[i]>(1633+offset2))
{
	if(input[i]==(1635+offset2))
{
	vR[i] = 1635;
}
else
{
	vR[i] = 1634;
}

}
else
{
	if(input[i]==(1633+offset2))
{
	vR[i] = 1633;
}
else
{
	vR[i] = 1632;
}

}

}

}

}

}
else
{
	if(input[i]>(1615+offset2))
{
	if(input[i]>(1623+offset2))
{
	if(input[i]>(1627+offset2))
{
	if(input[i]>(1629+offset2))
{
	if(input[i]==(1631+offset2))
{
	vR[i] = 1631;
}
else
{
	vR[i] = 1630;
}

}
else
{
	if(input[i]==(1629+offset2))
{
	vR[i] = 1629;
}
else
{
	vR[i] = 1628;
}

}

}
else
{
	if(input[i]>(1625+offset2))
{
	if(input[i]==(1627+offset2))
{
	vR[i] = 1627;
}
else
{
	vR[i] = 1626;
}

}
else
{
	if(input[i]==(1625+offset2))
{
	vR[i] = 1625;
}
else
{
	vR[i] = 1624;
}

}

}

}
else
{
	if(input[i]>(1619+offset2))
{
	if(input[i]>(1621+offset2))
{
	if(input[i]==(1623+offset2))
{
	vR[i] = 1623;
}
else
{
	vR[i] = 1622;
}

}
else
{
	if(input[i]==(1621+offset2))
{
	vR[i] = 1621;
}
else
{
	vR[i] = 1620;
}

}

}
else
{
	if(input[i]>(1617+offset2))
{
	if(input[i]==(1619+offset2))
{
	vR[i] = 1619;
}
else
{
	vR[i] = 1618;
}

}
else
{
	if(input[i]==(1617+offset2))
{
	vR[i] = 1617;
}
else
{
	vR[i] = 1616;
}

}

}

}

}
else
{
	if(input[i]>(1607+offset2))
{
	if(input[i]>(1611+offset2))
{
	if(input[i]>(1613+offset2))
{
	if(input[i]==(1615+offset2))
{
	vR[i] = 1615;
}
else
{
	vR[i] = 1614;
}

}
else
{
	if(input[i]==(1613+offset2))
{
	vR[i] = 1613;
}
else
{
	vR[i] = 1612;
}

}

}
else
{
	if(input[i]>(1609+offset2))
{
	if(input[i]==(1611+offset2))
{
	vR[i] = 1611;
}
else
{
	vR[i] = 1610;
}

}
else
{
	if(input[i]==(1609+offset2))
{
	vR[i] = 1609;
}
else
{
	vR[i] = 1608;
}

}

}

}
else
{
	if(input[i]>(1603+offset2))
{
	if(input[i]>(1605+offset2))
{
	if(input[i]==(1607+offset2))
{
	vR[i] = 1607;
}
else
{
	vR[i] = 1606;
}

}
else
{
	if(input[i]==(1605+offset2))
{
	vR[i] = 1605;
}
else
{
	vR[i] = 1604;
}

}

}
else
{
	if(input[i]>(1601+offset2))
{
	if(input[i]==(1603+offset2))
{
	vR[i] = 1603;
}
else
{
	vR[i] = 1602;
}

}
else
{
	if(input[i]==(1601+offset2))
{
	vR[i] = 1601;
}
else
{
	vR[i] = 1600;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1567+offset2))
{
	if(input[i]>(1583+offset2))
{
	if(input[i]>(1591+offset2))
{
	if(input[i]>(1595+offset2))
{
	if(input[i]>(1597+offset2))
{
	if(input[i]==(1599+offset2))
{
	vR[i] = 1599;
}
else
{
	vR[i] = 1598;
}

}
else
{
	if(input[i]==(1597+offset2))
{
	vR[i] = 1597;
}
else
{
	vR[i] = 1596;
}

}

}
else
{
	if(input[i]>(1593+offset2))
{
	if(input[i]==(1595+offset2))
{
	vR[i] = 1595;
}
else
{
	vR[i] = 1594;
}

}
else
{
	if(input[i]==(1593+offset2))
{
	vR[i] = 1593;
}
else
{
	vR[i] = 1592;
}

}

}

}
else
{
	if(input[i]>(1587+offset2))
{
	if(input[i]>(1589+offset2))
{
	if(input[i]==(1591+offset2))
{
	vR[i] = 1591;
}
else
{
	vR[i] = 1590;
}

}
else
{
	if(input[i]==(1589+offset2))
{
	vR[i] = 1589;
}
else
{
	vR[i] = 1588;
}

}

}
else
{
	if(input[i]>(1585+offset2))
{
	if(input[i]==(1587+offset2))
{
	vR[i] = 1587;
}
else
{
	vR[i] = 1586;
}

}
else
{
	if(input[i]==(1585+offset2))
{
	vR[i] = 1585;
}
else
{
	vR[i] = 1584;
}

}

}

}

}
else
{
	if(input[i]>(1575+offset2))
{
	if(input[i]>(1579+offset2))
{
	if(input[i]>(1581+offset2))
{
	if(input[i]==(1583+offset2))
{
	vR[i] = 1583;
}
else
{
	vR[i] = 1582;
}

}
else
{
	if(input[i]==(1581+offset2))
{
	vR[i] = 1581;
}
else
{
	vR[i] = 1580;
}

}

}
else
{
	if(input[i]>(1577+offset2))
{
	if(input[i]==(1579+offset2))
{
	vR[i] = 1579;
}
else
{
	vR[i] = 1578;
}

}
else
{
	if(input[i]==(1577+offset2))
{
	vR[i] = 1577;
}
else
{
	vR[i] = 1576;
}

}

}

}
else
{
	if(input[i]>(1571+offset2))
{
	if(input[i]>(1573+offset2))
{
	if(input[i]==(1575+offset2))
{
	vR[i] = 1575;
}
else
{
	vR[i] = 1574;
}

}
else
{
	if(input[i]==(1573+offset2))
{
	vR[i] = 1573;
}
else
{
	vR[i] = 1572;
}

}

}
else
{
	if(input[i]>(1569+offset2))
{
	if(input[i]==(1571+offset2))
{
	vR[i] = 1571;
}
else
{
	vR[i] = 1570;
}

}
else
{
	if(input[i]==(1569+offset2))
{
	vR[i] = 1569;
}
else
{
	vR[i] = 1568;
}

}

}

}

}

}
else
{
	if(input[i]>(1551+offset2))
{
	if(input[i]>(1559+offset2))
{
	if(input[i]>(1563+offset2))
{
	if(input[i]>(1565+offset2))
{
	if(input[i]==(1567+offset2))
{
	vR[i] = 1567;
}
else
{
	vR[i] = 1566;
}

}
else
{
	if(input[i]==(1565+offset2))
{
	vR[i] = 1565;
}
else
{
	vR[i] = 1564;
}

}

}
else
{
	if(input[i]>(1561+offset2))
{
	if(input[i]==(1563+offset2))
{
	vR[i] = 1563;
}
else
{
	vR[i] = 1562;
}

}
else
{
	if(input[i]==(1561+offset2))
{
	vR[i] = 1561;
}
else
{
	vR[i] = 1560;
}

}

}

}
else
{
	if(input[i]>(1555+offset2))
{
	if(input[i]>(1557+offset2))
{
	if(input[i]==(1559+offset2))
{
	vR[i] = 1559;
}
else
{
	vR[i] = 1558;
}

}
else
{
	if(input[i]==(1557+offset2))
{
	vR[i] = 1557;
}
else
{
	vR[i] = 1556;
}

}

}
else
{
	if(input[i]>(1553+offset2))
{
	if(input[i]==(1555+offset2))
{
	vR[i] = 1555;
}
else
{
	vR[i] = 1554;
}

}
else
{
	if(input[i]==(1553+offset2))
{
	vR[i] = 1553;
}
else
{
	vR[i] = 1552;
}

}

}

}

}
else
{
	if(input[i]>(1543+offset2))
{
	if(input[i]>(1547+offset2))
{
	if(input[i]>(1549+offset2))
{
	if(input[i]==(1551+offset2))
{
	vR[i] = 1551;
}
else
{
	vR[i] = 1550;
}

}
else
{
	if(input[i]==(1549+offset2))
{
	vR[i] = 1549;
}
else
{
	vR[i] = 1548;
}

}

}
else
{
	if(input[i]>(1545+offset2))
{
	if(input[i]==(1547+offset2))
{
	vR[i] = 1547;
}
else
{
	vR[i] = 1546;
}

}
else
{
	if(input[i]==(1545+offset2))
{
	vR[i] = 1545;
}
else
{
	vR[i] = 1544;
}

}

}

}
else
{
	if(input[i]>(1539+offset2))
{
	if(input[i]>(1541+offset2))
{
	if(input[i]==(1543+offset2))
{
	vR[i] = 1543;
}
else
{
	vR[i] = 1542;
}

}
else
{
	if(input[i]==(1541+offset2))
{
	vR[i] = 1541;
}
else
{
	vR[i] = 1540;
}

}

}
else
{
	if(input[i]>(1537+offset2))
{
	if(input[i]==(1539+offset2))
{
	vR[i] = 1539;
}
else
{
	vR[i] = 1538;
}

}
else
{
	if(input[i]==(1537+offset2))
{
	vR[i] = 1537;
}
else
{
	vR[i] = 1536;
}

}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(1279+offset2))
{
	if(input[i]>(1407+offset2))
{
	if(input[i]>(1471+offset2))
{
	if(input[i]>(1503+offset2))
{
	if(input[i]>(1519+offset2))
{
	if(input[i]>(1527+offset2))
{
	if(input[i]>(1531+offset2))
{
	if(input[i]>(1533+offset2))
{
	if(input[i]==(1535+offset2))
{
	vR[i] = 1535;
}
else
{
	vR[i] = 1534;
}

}
else
{
	if(input[i]==(1533+offset2))
{
	vR[i] = 1533;
}
else
{
	vR[i] = 1532;
}

}

}
else
{
	if(input[i]>(1529+offset2))
{
	if(input[i]==(1531+offset2))
{
	vR[i] = 1531;
}
else
{
	vR[i] = 1530;
}

}
else
{
	if(input[i]==(1529+offset2))
{
	vR[i] = 1529;
}
else
{
	vR[i] = 1528;
}

}

}

}
else
{
	if(input[i]>(1523+offset2))
{
	if(input[i]>(1525+offset2))
{
	if(input[i]==(1527+offset2))
{
	vR[i] = 1527;
}
else
{
	vR[i] = 1526;
}

}
else
{
	if(input[i]==(1525+offset2))
{
	vR[i] = 1525;
}
else
{
	vR[i] = 1524;
}

}

}
else
{
	if(input[i]>(1521+offset2))
{
	if(input[i]==(1523+offset2))
{
	vR[i] = 1523;
}
else
{
	vR[i] = 1522;
}

}
else
{
	if(input[i]==(1521+offset2))
{
	vR[i] = 1521;
}
else
{
	vR[i] = 1520;
}

}

}

}

}
else
{
	if(input[i]>(1511+offset2))
{
	if(input[i]>(1515+offset2))
{
	if(input[i]>(1517+offset2))
{
	if(input[i]==(1519+offset2))
{
	vR[i] = 1519;
}
else
{
	vR[i] = 1518;
}

}
else
{
	if(input[i]==(1517+offset2))
{
	vR[i] = 1517;
}
else
{
	vR[i] = 1516;
}

}

}
else
{
	if(input[i]>(1513+offset2))
{
	if(input[i]==(1515+offset2))
{
	vR[i] = 1515;
}
else
{
	vR[i] = 1514;
}

}
else
{
	if(input[i]==(1513+offset2))
{
	vR[i] = 1513;
}
else
{
	vR[i] = 1512;
}

}

}

}
else
{
	if(input[i]>(1507+offset2))
{
	if(input[i]>(1509+offset2))
{
	if(input[i]==(1511+offset2))
{
	vR[i] = 1511;
}
else
{
	vR[i] = 1510;
}

}
else
{
	if(input[i]==(1509+offset2))
{
	vR[i] = 1509;
}
else
{
	vR[i] = 1508;
}

}

}
else
{
	if(input[i]>(1505+offset2))
{
	if(input[i]==(1507+offset2))
{
	vR[i] = 1507;
}
else
{
	vR[i] = 1506;
}

}
else
{
	if(input[i]==(1505+offset2))
{
	vR[i] = 1505;
}
else
{
	vR[i] = 1504;
}

}

}

}

}

}
else
{
	if(input[i]>(1487+offset2))
{
	if(input[i]>(1495+offset2))
{
	if(input[i]>(1499+offset2))
{
	if(input[i]>(1501+offset2))
{
	if(input[i]==(1503+offset2))
{
	vR[i] = 1503;
}
else
{
	vR[i] = 1502;
}

}
else
{
	if(input[i]==(1501+offset2))
{
	vR[i] = 1501;
}
else
{
	vR[i] = 1500;
}

}

}
else
{
	if(input[i]>(1497+offset2))
{
	if(input[i]==(1499+offset2))
{
	vR[i] = 1499;
}
else
{
	vR[i] = 1498;
}

}
else
{
	if(input[i]==(1497+offset2))
{
	vR[i] = 1497;
}
else
{
	vR[i] = 1496;
}

}

}

}
else
{
	if(input[i]>(1491+offset2))
{
	if(input[i]>(1493+offset2))
{
	if(input[i]==(1495+offset2))
{
	vR[i] = 1495;
}
else
{
	vR[i] = 1494;
}

}
else
{
	if(input[i]==(1493+offset2))
{
	vR[i] = 1493;
}
else
{
	vR[i] = 1492;
}

}

}
else
{
	if(input[i]>(1489+offset2))
{
	if(input[i]==(1491+offset2))
{
	vR[i] = 1491;
}
else
{
	vR[i] = 1490;
}

}
else
{
	if(input[i]==(1489+offset2))
{
	vR[i] = 1489;
}
else
{
	vR[i] = 1488;
}

}

}

}

}
else
{
	if(input[i]>(1479+offset2))
{
	if(input[i]>(1483+offset2))
{
	if(input[i]>(1485+offset2))
{
	if(input[i]==(1487+offset2))
{
	vR[i] = 1487;
}
else
{
	vR[i] = 1486;
}

}
else
{
	if(input[i]==(1485+offset2))
{
	vR[i] = 1485;
}
else
{
	vR[i] = 1484;
}

}

}
else
{
	if(input[i]>(1481+offset2))
{
	if(input[i]==(1483+offset2))
{
	vR[i] = 1483;
}
else
{
	vR[i] = 1482;
}

}
else
{
	if(input[i]==(1481+offset2))
{
	vR[i] = 1481;
}
else
{
	vR[i] = 1480;
}

}

}

}
else
{
	if(input[i]>(1475+offset2))
{
	if(input[i]>(1477+offset2))
{
	if(input[i]==(1479+offset2))
{
	vR[i] = 1479;
}
else
{
	vR[i] = 1478;
}

}
else
{
	if(input[i]==(1477+offset2))
{
	vR[i] = 1477;
}
else
{
	vR[i] = 1476;
}

}

}
else
{
	if(input[i]>(1473+offset2))
{
	if(input[i]==(1475+offset2))
{
	vR[i] = 1475;
}
else
{
	vR[i] = 1474;
}

}
else
{
	if(input[i]==(1473+offset2))
{
	vR[i] = 1473;
}
else
{
	vR[i] = 1472;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1439+offset2))
{
	if(input[i]>(1455+offset2))
{
	if(input[i]>(1463+offset2))
{
	if(input[i]>(1467+offset2))
{
	if(input[i]>(1469+offset2))
{
	if(input[i]==(1471+offset2))
{
	vR[i] = 1471;
}
else
{
	vR[i] = 1470;
}

}
else
{
	if(input[i]==(1469+offset2))
{
	vR[i] = 1469;
}
else
{
	vR[i] = 1468;
}

}

}
else
{
	if(input[i]>(1465+offset2))
{
	if(input[i]==(1467+offset2))
{
	vR[i] = 1467;
}
else
{
	vR[i] = 1466;
}

}
else
{
	if(input[i]==(1465+offset2))
{
	vR[i] = 1465;
}
else
{
	vR[i] = 1464;
}

}

}

}
else
{
	if(input[i]>(1459+offset2))
{
	if(input[i]>(1461+offset2))
{
	if(input[i]==(1463+offset2))
{
	vR[i] = 1463;
}
else
{
	vR[i] = 1462;
}

}
else
{
	if(input[i]==(1461+offset2))
{
	vR[i] = 1461;
}
else
{
	vR[i] = 1460;
}

}

}
else
{
	if(input[i]>(1457+offset2))
{
	if(input[i]==(1459+offset2))
{
	vR[i] = 1459;
}
else
{
	vR[i] = 1458;
}

}
else
{
	if(input[i]==(1457+offset2))
{
	vR[i] = 1457;
}
else
{
	vR[i] = 1456;
}

}

}

}

}
else
{
	if(input[i]>(1447+offset2))
{
	if(input[i]>(1451+offset2))
{
	if(input[i]>(1453+offset2))
{
	if(input[i]==(1455+offset2))
{
	vR[i] = 1455;
}
else
{
	vR[i] = 1454;
}

}
else
{
	if(input[i]==(1453+offset2))
{
	vR[i] = 1453;
}
else
{
	vR[i] = 1452;
}

}

}
else
{
	if(input[i]>(1449+offset2))
{
	if(input[i]==(1451+offset2))
{
	vR[i] = 1451;
}
else
{
	vR[i] = 1450;
}

}
else
{
	if(input[i]==(1449+offset2))
{
	vR[i] = 1449;
}
else
{
	vR[i] = 1448;
}

}

}

}
else
{
	if(input[i]>(1443+offset2))
{
	if(input[i]>(1445+offset2))
{
	if(input[i]==(1447+offset2))
{
	vR[i] = 1447;
}
else
{
	vR[i] = 1446;
}

}
else
{
	if(input[i]==(1445+offset2))
{
	vR[i] = 1445;
}
else
{
	vR[i] = 1444;
}

}

}
else
{
	if(input[i]>(1441+offset2))
{
	if(input[i]==(1443+offset2))
{
	vR[i] = 1443;
}
else
{
	vR[i] = 1442;
}

}
else
{
	if(input[i]==(1441+offset2))
{
	vR[i] = 1441;
}
else
{
	vR[i] = 1440;
}

}

}

}

}

}
else
{
	if(input[i]>(1423+offset2))
{
	if(input[i]>(1431+offset2))
{
	if(input[i]>(1435+offset2))
{
	if(input[i]>(1437+offset2))
{
	if(input[i]==(1439+offset2))
{
	vR[i] = 1439;
}
else
{
	vR[i] = 1438;
}

}
else
{
	if(input[i]==(1437+offset2))
{
	vR[i] = 1437;
}
else
{
	vR[i] = 1436;
}

}

}
else
{
	if(input[i]>(1433+offset2))
{
	if(input[i]==(1435+offset2))
{
	vR[i] = 1435;
}
else
{
	vR[i] = 1434;
}

}
else
{
	if(input[i]==(1433+offset2))
{
	vR[i] = 1433;
}
else
{
	vR[i] = 1432;
}

}

}

}
else
{
	if(input[i]>(1427+offset2))
{
	if(input[i]>(1429+offset2))
{
	if(input[i]==(1431+offset2))
{
	vR[i] = 1431;
}
else
{
	vR[i] = 1430;
}

}
else
{
	if(input[i]==(1429+offset2))
{
	vR[i] = 1429;
}
else
{
	vR[i] = 1428;
}

}

}
else
{
	if(input[i]>(1425+offset2))
{
	if(input[i]==(1427+offset2))
{
	vR[i] = 1427;
}
else
{
	vR[i] = 1426;
}

}
else
{
	if(input[i]==(1425+offset2))
{
	vR[i] = 1425;
}
else
{
	vR[i] = 1424;
}

}

}

}

}
else
{
	if(input[i]>(1415+offset2))
{
	if(input[i]>(1419+offset2))
{
	if(input[i]>(1421+offset2))
{
	if(input[i]==(1423+offset2))
{
	vR[i] = 1423;
}
else
{
	vR[i] = 1422;
}

}
else
{
	if(input[i]==(1421+offset2))
{
	vR[i] = 1421;
}
else
{
	vR[i] = 1420;
}

}

}
else
{
	if(input[i]>(1417+offset2))
{
	if(input[i]==(1419+offset2))
{
	vR[i] = 1419;
}
else
{
	vR[i] = 1418;
}

}
else
{
	if(input[i]==(1417+offset2))
{
	vR[i] = 1417;
}
else
{
	vR[i] = 1416;
}

}

}

}
else
{
	if(input[i]>(1411+offset2))
{
	if(input[i]>(1413+offset2))
{
	if(input[i]==(1415+offset2))
{
	vR[i] = 1415;
}
else
{
	vR[i] = 1414;
}

}
else
{
	if(input[i]==(1413+offset2))
{
	vR[i] = 1413;
}
else
{
	vR[i] = 1412;
}

}

}
else
{
	if(input[i]>(1409+offset2))
{
	if(input[i]==(1411+offset2))
{
	vR[i] = 1411;
}
else
{
	vR[i] = 1410;
}

}
else
{
	if(input[i]==(1409+offset2))
{
	vR[i] = 1409;
}
else
{
	vR[i] = 1408;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(1343+offset2))
{
	if(input[i]>(1375+offset2))
{
	if(input[i]>(1391+offset2))
{
	if(input[i]>(1399+offset2))
{
	if(input[i]>(1403+offset2))
{
	if(input[i]>(1405+offset2))
{
	if(input[i]==(1407+offset2))
{
	vR[i] = 1407;
}
else
{
	vR[i] = 1406;
}

}
else
{
	if(input[i]==(1405+offset2))
{
	vR[i] = 1405;
}
else
{
	vR[i] = 1404;
}

}

}
else
{
	if(input[i]>(1401+offset2))
{
	if(input[i]==(1403+offset2))
{
	vR[i] = 1403;
}
else
{
	vR[i] = 1402;
}

}
else
{
	if(input[i]==(1401+offset2))
{
	vR[i] = 1401;
}
else
{
	vR[i] = 1400;
}

}

}

}
else
{
	if(input[i]>(1395+offset2))
{
	if(input[i]>(1397+offset2))
{
	if(input[i]==(1399+offset2))
{
	vR[i] = 1399;
}
else
{
	vR[i] = 1398;
}

}
else
{
	if(input[i]==(1397+offset2))
{
	vR[i] = 1397;
}
else
{
	vR[i] = 1396;
}

}

}
else
{
	if(input[i]>(1393+offset2))
{
	if(input[i]==(1395+offset2))
{
	vR[i] = 1395;
}
else
{
	vR[i] = 1394;
}

}
else
{
	if(input[i]==(1393+offset2))
{
	vR[i] = 1393;
}
else
{
	vR[i] = 1392;
}

}

}

}

}
else
{
	if(input[i]>(1383+offset2))
{
	if(input[i]>(1387+offset2))
{
	if(input[i]>(1389+offset2))
{
	if(input[i]==(1391+offset2))
{
	vR[i] = 1391;
}
else
{
	vR[i] = 1390;
}

}
else
{
	if(input[i]==(1389+offset2))
{
	vR[i] = 1389;
}
else
{
	vR[i] = 1388;
}

}

}
else
{
	if(input[i]>(1385+offset2))
{
	if(input[i]==(1387+offset2))
{
	vR[i] = 1387;
}
else
{
	vR[i] = 1386;
}

}
else
{
	if(input[i]==(1385+offset2))
{
	vR[i] = 1385;
}
else
{
	vR[i] = 1384;
}

}

}

}
else
{
	if(input[i]>(1379+offset2))
{
	if(input[i]>(1381+offset2))
{
	if(input[i]==(1383+offset2))
{
	vR[i] = 1383;
}
else
{
	vR[i] = 1382;
}

}
else
{
	if(input[i]==(1381+offset2))
{
	vR[i] = 1381;
}
else
{
	vR[i] = 1380;
}

}

}
else
{
	if(input[i]>(1377+offset2))
{
	if(input[i]==(1379+offset2))
{
	vR[i] = 1379;
}
else
{
	vR[i] = 1378;
}

}
else
{
	if(input[i]==(1377+offset2))
{
	vR[i] = 1377;
}
else
{
	vR[i] = 1376;
}

}

}

}

}

}
else
{
	if(input[i]>(1359+offset2))
{
	if(input[i]>(1367+offset2))
{
	if(input[i]>(1371+offset2))
{
	if(input[i]>(1373+offset2))
{
	if(input[i]==(1375+offset2))
{
	vR[i] = 1375;
}
else
{
	vR[i] = 1374;
}

}
else
{
	if(input[i]==(1373+offset2))
{
	vR[i] = 1373;
}
else
{
	vR[i] = 1372;
}

}

}
else
{
	if(input[i]>(1369+offset2))
{
	if(input[i]==(1371+offset2))
{
	vR[i] = 1371;
}
else
{
	vR[i] = 1370;
}

}
else
{
	if(input[i]==(1369+offset2))
{
	vR[i] = 1369;
}
else
{
	vR[i] = 1368;
}

}

}

}
else
{
	if(input[i]>(1363+offset2))
{
	if(input[i]>(1365+offset2))
{
	if(input[i]==(1367+offset2))
{
	vR[i] = 1367;
}
else
{
	vR[i] = 1366;
}

}
else
{
	if(input[i]==(1365+offset2))
{
	vR[i] = 1365;
}
else
{
	vR[i] = 1364;
}

}

}
else
{
	if(input[i]>(1361+offset2))
{
	if(input[i]==(1363+offset2))
{
	vR[i] = 1363;
}
else
{
	vR[i] = 1362;
}

}
else
{
	if(input[i]==(1361+offset2))
{
	vR[i] = 1361;
}
else
{
	vR[i] = 1360;
}

}

}

}

}
else
{
	if(input[i]>(1351+offset2))
{
	if(input[i]>(1355+offset2))
{
	if(input[i]>(1357+offset2))
{
	if(input[i]==(1359+offset2))
{
	vR[i] = 1359;
}
else
{
	vR[i] = 1358;
}

}
else
{
	if(input[i]==(1357+offset2))
{
	vR[i] = 1357;
}
else
{
	vR[i] = 1356;
}

}

}
else
{
	if(input[i]>(1353+offset2))
{
	if(input[i]==(1355+offset2))
{
	vR[i] = 1355;
}
else
{
	vR[i] = 1354;
}

}
else
{
	if(input[i]==(1353+offset2))
{
	vR[i] = 1353;
}
else
{
	vR[i] = 1352;
}

}

}

}
else
{
	if(input[i]>(1347+offset2))
{
	if(input[i]>(1349+offset2))
{
	if(input[i]==(1351+offset2))
{
	vR[i] = 1351;
}
else
{
	vR[i] = 1350;
}

}
else
{
	if(input[i]==(1349+offset2))
{
	vR[i] = 1349;
}
else
{
	vR[i] = 1348;
}

}

}
else
{
	if(input[i]>(1345+offset2))
{
	if(input[i]==(1347+offset2))
{
	vR[i] = 1347;
}
else
{
	vR[i] = 1346;
}

}
else
{
	if(input[i]==(1345+offset2))
{
	vR[i] = 1345;
}
else
{
	vR[i] = 1344;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1311+offset2))
{
	if(input[i]>(1327+offset2))
{
	if(input[i]>(1335+offset2))
{
	if(input[i]>(1339+offset2))
{
	if(input[i]>(1341+offset2))
{
	if(input[i]==(1343+offset2))
{
	vR[i] = 1343;
}
else
{
	vR[i] = 1342;
}

}
else
{
	if(input[i]==(1341+offset2))
{
	vR[i] = 1341;
}
else
{
	vR[i] = 1340;
}

}

}
else
{
	if(input[i]>(1337+offset2))
{
	if(input[i]==(1339+offset2))
{
	vR[i] = 1339;
}
else
{
	vR[i] = 1338;
}

}
else
{
	if(input[i]==(1337+offset2))
{
	vR[i] = 1337;
}
else
{
	vR[i] = 1336;
}

}

}

}
else
{
	if(input[i]>(1331+offset2))
{
	if(input[i]>(1333+offset2))
{
	if(input[i]==(1335+offset2))
{
	vR[i] = 1335;
}
else
{
	vR[i] = 1334;
}

}
else
{
	if(input[i]==(1333+offset2))
{
	vR[i] = 1333;
}
else
{
	vR[i] = 1332;
}

}

}
else
{
	if(input[i]>(1329+offset2))
{
	if(input[i]==(1331+offset2))
{
	vR[i] = 1331;
}
else
{
	vR[i] = 1330;
}

}
else
{
	if(input[i]==(1329+offset2))
{
	vR[i] = 1329;
}
else
{
	vR[i] = 1328;
}

}

}

}

}
else
{
	if(input[i]>(1319+offset2))
{
	if(input[i]>(1323+offset2))
{
	if(input[i]>(1325+offset2))
{
	if(input[i]==(1327+offset2))
{
	vR[i] = 1327;
}
else
{
	vR[i] = 1326;
}

}
else
{
	if(input[i]==(1325+offset2))
{
	vR[i] = 1325;
}
else
{
	vR[i] = 1324;
}

}

}
else
{
	if(input[i]>(1321+offset2))
{
	if(input[i]==(1323+offset2))
{
	vR[i] = 1323;
}
else
{
	vR[i] = 1322;
}

}
else
{
	if(input[i]==(1321+offset2))
{
	vR[i] = 1321;
}
else
{
	vR[i] = 1320;
}

}

}

}
else
{
	if(input[i]>(1315+offset2))
{
	if(input[i]>(1317+offset2))
{
	if(input[i]==(1319+offset2))
{
	vR[i] = 1319;
}
else
{
	vR[i] = 1318;
}

}
else
{
	if(input[i]==(1317+offset2))
{
	vR[i] = 1317;
}
else
{
	vR[i] = 1316;
}

}

}
else
{
	if(input[i]>(1313+offset2))
{
	if(input[i]==(1315+offset2))
{
	vR[i] = 1315;
}
else
{
	vR[i] = 1314;
}

}
else
{
	if(input[i]==(1313+offset2))
{
	vR[i] = 1313;
}
else
{
	vR[i] = 1312;
}

}

}

}

}

}
else
{
	if(input[i]>(1295+offset2))
{
	if(input[i]>(1303+offset2))
{
	if(input[i]>(1307+offset2))
{
	if(input[i]>(1309+offset2))
{
	if(input[i]==(1311+offset2))
{
	vR[i] = 1311;
}
else
{
	vR[i] = 1310;
}

}
else
{
	if(input[i]==(1309+offset2))
{
	vR[i] = 1309;
}
else
{
	vR[i] = 1308;
}

}

}
else
{
	if(input[i]>(1305+offset2))
{
	if(input[i]==(1307+offset2))
{
	vR[i] = 1307;
}
else
{
	vR[i] = 1306;
}

}
else
{
	if(input[i]==(1305+offset2))
{
	vR[i] = 1305;
}
else
{
	vR[i] = 1304;
}

}

}

}
else
{
	if(input[i]>(1299+offset2))
{
	if(input[i]>(1301+offset2))
{
	if(input[i]==(1303+offset2))
{
	vR[i] = 1303;
}
else
{
	vR[i] = 1302;
}

}
else
{
	if(input[i]==(1301+offset2))
{
	vR[i] = 1301;
}
else
{
	vR[i] = 1300;
}

}

}
else
{
	if(input[i]>(1297+offset2))
{
	if(input[i]==(1299+offset2))
{
	vR[i] = 1299;
}
else
{
	vR[i] = 1298;
}

}
else
{
	if(input[i]==(1297+offset2))
{
	vR[i] = 1297;
}
else
{
	vR[i] = 1296;
}

}

}

}

}
else
{
	if(input[i]>(1287+offset2))
{
	if(input[i]>(1291+offset2))
{
	if(input[i]>(1293+offset2))
{
	if(input[i]==(1295+offset2))
{
	vR[i] = 1295;
}
else
{
	vR[i] = 1294;
}

}
else
{
	if(input[i]==(1293+offset2))
{
	vR[i] = 1293;
}
else
{
	vR[i] = 1292;
}

}

}
else
{
	if(input[i]>(1289+offset2))
{
	if(input[i]==(1291+offset2))
{
	vR[i] = 1291;
}
else
{
	vR[i] = 1290;
}

}
else
{
	if(input[i]==(1289+offset2))
{
	vR[i] = 1289;
}
else
{
	vR[i] = 1288;
}

}

}

}
else
{
	if(input[i]>(1283+offset2))
{
	if(input[i]>(1285+offset2))
{
	if(input[i]==(1287+offset2))
{
	vR[i] = 1287;
}
else
{
	vR[i] = 1286;
}

}
else
{
	if(input[i]==(1285+offset2))
{
	vR[i] = 1285;
}
else
{
	vR[i] = 1284;
}

}

}
else
{
	if(input[i]>(1281+offset2))
{
	if(input[i]==(1283+offset2))
{
	vR[i] = 1283;
}
else
{
	vR[i] = 1282;
}

}
else
{
	if(input[i]==(1281+offset2))
{
	vR[i] = 1281;
}
else
{
	vR[i] = 1280;
}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(1151+offset2))
{
	if(input[i]>(1215+offset2))
{
	if(input[i]>(1247+offset2))
{
	if(input[i]>(1263+offset2))
{
	if(input[i]>(1271+offset2))
{
	if(input[i]>(1275+offset2))
{
	if(input[i]>(1277+offset2))
{
	if(input[i]==(1279+offset2))
{
	vR[i] = 1279;
}
else
{
	vR[i] = 1278;
}

}
else
{
	if(input[i]==(1277+offset2))
{
	vR[i] = 1277;
}
else
{
	vR[i] = 1276;
}

}

}
else
{
	if(input[i]>(1273+offset2))
{
	if(input[i]==(1275+offset2))
{
	vR[i] = 1275;
}
else
{
	vR[i] = 1274;
}

}
else
{
	if(input[i]==(1273+offset2))
{
	vR[i] = 1273;
}
else
{
	vR[i] = 1272;
}

}

}

}
else
{
	if(input[i]>(1267+offset2))
{
	if(input[i]>(1269+offset2))
{
	if(input[i]==(1271+offset2))
{
	vR[i] = 1271;
}
else
{
	vR[i] = 1270;
}

}
else
{
	if(input[i]==(1269+offset2))
{
	vR[i] = 1269;
}
else
{
	vR[i] = 1268;
}

}

}
else
{
	if(input[i]>(1265+offset2))
{
	if(input[i]==(1267+offset2))
{
	vR[i] = 1267;
}
else
{
	vR[i] = 1266;
}

}
else
{
	if(input[i]==(1265+offset2))
{
	vR[i] = 1265;
}
else
{
	vR[i] = 1264;
}

}

}

}

}
else
{
	if(input[i]>(1255+offset2))
{
	if(input[i]>(1259+offset2))
{
	if(input[i]>(1261+offset2))
{
	if(input[i]==(1263+offset2))
{
	vR[i] = 1263;
}
else
{
	vR[i] = 1262;
}

}
else
{
	if(input[i]==(1261+offset2))
{
	vR[i] = 1261;
}
else
{
	vR[i] = 1260;
}

}

}
else
{
	if(input[i]>(1257+offset2))
{
	if(input[i]==(1259+offset2))
{
	vR[i] = 1259;
}
else
{
	vR[i] = 1258;
}

}
else
{
	if(input[i]==(1257+offset2))
{
	vR[i] = 1257;
}
else
{
	vR[i] = 1256;
}

}

}

}
else
{
	if(input[i]>(1251+offset2))
{
	if(input[i]>(1253+offset2))
{
	if(input[i]==(1255+offset2))
{
	vR[i] = 1255;
}
else
{
	vR[i] = 1254;
}

}
else
{
	if(input[i]==(1253+offset2))
{
	vR[i] = 1253;
}
else
{
	vR[i] = 1252;
}

}

}
else
{
	if(input[i]>(1249+offset2))
{
	if(input[i]==(1251+offset2))
{
	vR[i] = 1251;
}
else
{
	vR[i] = 1250;
}

}
else
{
	if(input[i]==(1249+offset2))
{
	vR[i] = 1249;
}
else
{
	vR[i] = 1248;
}

}

}

}

}

}
else
{
	if(input[i]>(1231+offset2))
{
	if(input[i]>(1239+offset2))
{
	if(input[i]>(1243+offset2))
{
	if(input[i]>(1245+offset2))
{
	if(input[i]==(1247+offset2))
{
	vR[i] = 1247;
}
else
{
	vR[i] = 1246;
}

}
else
{
	if(input[i]==(1245+offset2))
{
	vR[i] = 1245;
}
else
{
	vR[i] = 1244;
}

}

}
else
{
	if(input[i]>(1241+offset2))
{
	if(input[i]==(1243+offset2))
{
	vR[i] = 1243;
}
else
{
	vR[i] = 1242;
}

}
else
{
	if(input[i]==(1241+offset2))
{
	vR[i] = 1241;
}
else
{
	vR[i] = 1240;
}

}

}

}
else
{
	if(input[i]>(1235+offset2))
{
	if(input[i]>(1237+offset2))
{
	if(input[i]==(1239+offset2))
{
	vR[i] = 1239;
}
else
{
	vR[i] = 1238;
}

}
else
{
	if(input[i]==(1237+offset2))
{
	vR[i] = 1237;
}
else
{
	vR[i] = 1236;
}

}

}
else
{
	if(input[i]>(1233+offset2))
{
	if(input[i]==(1235+offset2))
{
	vR[i] = 1235;
}
else
{
	vR[i] = 1234;
}

}
else
{
	if(input[i]==(1233+offset2))
{
	vR[i] = 1233;
}
else
{
	vR[i] = 1232;
}

}

}

}

}
else
{
	if(input[i]>(1223+offset2))
{
	if(input[i]>(1227+offset2))
{
	if(input[i]>(1229+offset2))
{
	if(input[i]==(1231+offset2))
{
	vR[i] = 1231;
}
else
{
	vR[i] = 1230;
}

}
else
{
	if(input[i]==(1229+offset2))
{
	vR[i] = 1229;
}
else
{
	vR[i] = 1228;
}

}

}
else
{
	if(input[i]>(1225+offset2))
{
	if(input[i]==(1227+offset2))
{
	vR[i] = 1227;
}
else
{
	vR[i] = 1226;
}

}
else
{
	if(input[i]==(1225+offset2))
{
	vR[i] = 1225;
}
else
{
	vR[i] = 1224;
}

}

}

}
else
{
	if(input[i]>(1219+offset2))
{
	if(input[i]>(1221+offset2))
{
	if(input[i]==(1223+offset2))
{
	vR[i] = 1223;
}
else
{
	vR[i] = 1222;
}

}
else
{
	if(input[i]==(1221+offset2))
{
	vR[i] = 1221;
}
else
{
	vR[i] = 1220;
}

}

}
else
{
	if(input[i]>(1217+offset2))
{
	if(input[i]==(1219+offset2))
{
	vR[i] = 1219;
}
else
{
	vR[i] = 1218;
}

}
else
{
	if(input[i]==(1217+offset2))
{
	vR[i] = 1217;
}
else
{
	vR[i] = 1216;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1183+offset2))
{
	if(input[i]>(1199+offset2))
{
	if(input[i]>(1207+offset2))
{
	if(input[i]>(1211+offset2))
{
	if(input[i]>(1213+offset2))
{
	if(input[i]==(1215+offset2))
{
	vR[i] = 1215;
}
else
{
	vR[i] = 1214;
}

}
else
{
	if(input[i]==(1213+offset2))
{
	vR[i] = 1213;
}
else
{
	vR[i] = 1212;
}

}

}
else
{
	if(input[i]>(1209+offset2))
{
	if(input[i]==(1211+offset2))
{
	vR[i] = 1211;
}
else
{
	vR[i] = 1210;
}

}
else
{
	if(input[i]==(1209+offset2))
{
	vR[i] = 1209;
}
else
{
	vR[i] = 1208;
}

}

}

}
else
{
	if(input[i]>(1203+offset2))
{
	if(input[i]>(1205+offset2))
{
	if(input[i]==(1207+offset2))
{
	vR[i] = 1207;
}
else
{
	vR[i] = 1206;
}

}
else
{
	if(input[i]==(1205+offset2))
{
	vR[i] = 1205;
}
else
{
	vR[i] = 1204;
}

}

}
else
{
	if(input[i]>(1201+offset2))
{
	if(input[i]==(1203+offset2))
{
	vR[i] = 1203;
}
else
{
	vR[i] = 1202;
}

}
else
{
	if(input[i]==(1201+offset2))
{
	vR[i] = 1201;
}
else
{
	vR[i] = 1200;
}

}

}

}

}
else
{
	if(input[i]>(1191+offset2))
{
	if(input[i]>(1195+offset2))
{
	if(input[i]>(1197+offset2))
{
	if(input[i]==(1199+offset2))
{
	vR[i] = 1199;
}
else
{
	vR[i] = 1198;
}

}
else
{
	if(input[i]==(1197+offset2))
{
	vR[i] = 1197;
}
else
{
	vR[i] = 1196;
}

}

}
else
{
	if(input[i]>(1193+offset2))
{
	if(input[i]==(1195+offset2))
{
	vR[i] = 1195;
}
else
{
	vR[i] = 1194;
}

}
else
{
	if(input[i]==(1193+offset2))
{
	vR[i] = 1193;
}
else
{
	vR[i] = 1192;
}

}

}

}
else
{
	if(input[i]>(1187+offset2))
{
	if(input[i]>(1189+offset2))
{
	if(input[i]==(1191+offset2))
{
	vR[i] = 1191;
}
else
{
	vR[i] = 1190;
}

}
else
{
	if(input[i]==(1189+offset2))
{
	vR[i] = 1189;
}
else
{
	vR[i] = 1188;
}

}

}
else
{
	if(input[i]>(1185+offset2))
{
	if(input[i]==(1187+offset2))
{
	vR[i] = 1187;
}
else
{
	vR[i] = 1186;
}

}
else
{
	if(input[i]==(1185+offset2))
{
	vR[i] = 1185;
}
else
{
	vR[i] = 1184;
}

}

}

}

}

}
else
{
	if(input[i]>(1167+offset2))
{
	if(input[i]>(1175+offset2))
{
	if(input[i]>(1179+offset2))
{
	if(input[i]>(1181+offset2))
{
	if(input[i]==(1183+offset2))
{
	vR[i] = 1183;
}
else
{
	vR[i] = 1182;
}

}
else
{
	if(input[i]==(1181+offset2))
{
	vR[i] = 1181;
}
else
{
	vR[i] = 1180;
}

}

}
else
{
	if(input[i]>(1177+offset2))
{
	if(input[i]==(1179+offset2))
{
	vR[i] = 1179;
}
else
{
	vR[i] = 1178;
}

}
else
{
	if(input[i]==(1177+offset2))
{
	vR[i] = 1177;
}
else
{
	vR[i] = 1176;
}

}

}

}
else
{
	if(input[i]>(1171+offset2))
{
	if(input[i]>(1173+offset2))
{
	if(input[i]==(1175+offset2))
{
	vR[i] = 1175;
}
else
{
	vR[i] = 1174;
}

}
else
{
	if(input[i]==(1173+offset2))
{
	vR[i] = 1173;
}
else
{
	vR[i] = 1172;
}

}

}
else
{
	if(input[i]>(1169+offset2))
{
	if(input[i]==(1171+offset2))
{
	vR[i] = 1171;
}
else
{
	vR[i] = 1170;
}

}
else
{
	if(input[i]==(1169+offset2))
{
	vR[i] = 1169;
}
else
{
	vR[i] = 1168;
}

}

}

}

}
else
{
	if(input[i]>(1159+offset2))
{
	if(input[i]>(1163+offset2))
{
	if(input[i]>(1165+offset2))
{
	if(input[i]==(1167+offset2))
{
	vR[i] = 1167;
}
else
{
	vR[i] = 1166;
}

}
else
{
	if(input[i]==(1165+offset2))
{
	vR[i] = 1165;
}
else
{
	vR[i] = 1164;
}

}

}
else
{
	if(input[i]>(1161+offset2))
{
	if(input[i]==(1163+offset2))
{
	vR[i] = 1163;
}
else
{
	vR[i] = 1162;
}

}
else
{
	if(input[i]==(1161+offset2))
{
	vR[i] = 1161;
}
else
{
	vR[i] = 1160;
}

}

}

}
else
{
	if(input[i]>(1155+offset2))
{
	if(input[i]>(1157+offset2))
{
	if(input[i]==(1159+offset2))
{
	vR[i] = 1159;
}
else
{
	vR[i] = 1158;
}

}
else
{
	if(input[i]==(1157+offset2))
{
	vR[i] = 1157;
}
else
{
	vR[i] = 1156;
}

}

}
else
{
	if(input[i]>(1153+offset2))
{
	if(input[i]==(1155+offset2))
{
	vR[i] = 1155;
}
else
{
	vR[i] = 1154;
}

}
else
{
	if(input[i]==(1153+offset2))
{
	vR[i] = 1153;
}
else
{
	vR[i] = 1152;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(1087+offset2))
{
	if(input[i]>(1119+offset2))
{
	if(input[i]>(1135+offset2))
{
	if(input[i]>(1143+offset2))
{
	if(input[i]>(1147+offset2))
{
	if(input[i]>(1149+offset2))
{
	if(input[i]==(1151+offset2))
{
	vR[i] = 1151;
}
else
{
	vR[i] = 1150;
}

}
else
{
	if(input[i]==(1149+offset2))
{
	vR[i] = 1149;
}
else
{
	vR[i] = 1148;
}

}

}
else
{
	if(input[i]>(1145+offset2))
{
	if(input[i]==(1147+offset2))
{
	vR[i] = 1147;
}
else
{
	vR[i] = 1146;
}

}
else
{
	if(input[i]==(1145+offset2))
{
	vR[i] = 1145;
}
else
{
	vR[i] = 1144;
}

}

}

}
else
{
	if(input[i]>(1139+offset2))
{
	if(input[i]>(1141+offset2))
{
	if(input[i]==(1143+offset2))
{
	vR[i] = 1143;
}
else
{
	vR[i] = 1142;
}

}
else
{
	if(input[i]==(1141+offset2))
{
	vR[i] = 1141;
}
else
{
	vR[i] = 1140;
}

}

}
else
{
	if(input[i]>(1137+offset2))
{
	if(input[i]==(1139+offset2))
{
	vR[i] = 1139;
}
else
{
	vR[i] = 1138;
}

}
else
{
	if(input[i]==(1137+offset2))
{
	vR[i] = 1137;
}
else
{
	vR[i] = 1136;
}

}

}

}

}
else
{
	if(input[i]>(1127+offset2))
{
	if(input[i]>(1131+offset2))
{
	if(input[i]>(1133+offset2))
{
	if(input[i]==(1135+offset2))
{
	vR[i] = 1135;
}
else
{
	vR[i] = 1134;
}

}
else
{
	if(input[i]==(1133+offset2))
{
	vR[i] = 1133;
}
else
{
	vR[i] = 1132;
}

}

}
else
{
	if(input[i]>(1129+offset2))
{
	if(input[i]==(1131+offset2))
{
	vR[i] = 1131;
}
else
{
	vR[i] = 1130;
}

}
else
{
	if(input[i]==(1129+offset2))
{
	vR[i] = 1129;
}
else
{
	vR[i] = 1128;
}

}

}

}
else
{
	if(input[i]>(1123+offset2))
{
	if(input[i]>(1125+offset2))
{
	if(input[i]==(1127+offset2))
{
	vR[i] = 1127;
}
else
{
	vR[i] = 1126;
}

}
else
{
	if(input[i]==(1125+offset2))
{
	vR[i] = 1125;
}
else
{
	vR[i] = 1124;
}

}

}
else
{
	if(input[i]>(1121+offset2))
{
	if(input[i]==(1123+offset2))
{
	vR[i] = 1123;
}
else
{
	vR[i] = 1122;
}

}
else
{
	if(input[i]==(1121+offset2))
{
	vR[i] = 1121;
}
else
{
	vR[i] = 1120;
}

}

}

}

}

}
else
{
	if(input[i]>(1103+offset2))
{
	if(input[i]>(1111+offset2))
{
	if(input[i]>(1115+offset2))
{
	if(input[i]>(1117+offset2))
{
	if(input[i]==(1119+offset2))
{
	vR[i] = 1119;
}
else
{
	vR[i] = 1118;
}

}
else
{
	if(input[i]==(1117+offset2))
{
	vR[i] = 1117;
}
else
{
	vR[i] = 1116;
}

}

}
else
{
	if(input[i]>(1113+offset2))
{
	if(input[i]==(1115+offset2))
{
	vR[i] = 1115;
}
else
{
	vR[i] = 1114;
}

}
else
{
	if(input[i]==(1113+offset2))
{
	vR[i] = 1113;
}
else
{
	vR[i] = 1112;
}

}

}

}
else
{
	if(input[i]>(1107+offset2))
{
	if(input[i]>(1109+offset2))
{
	if(input[i]==(1111+offset2))
{
	vR[i] = 1111;
}
else
{
	vR[i] = 1110;
}

}
else
{
	if(input[i]==(1109+offset2))
{
	vR[i] = 1109;
}
else
{
	vR[i] = 1108;
}

}

}
else
{
	if(input[i]>(1105+offset2))
{
	if(input[i]==(1107+offset2))
{
	vR[i] = 1107;
}
else
{
	vR[i] = 1106;
}

}
else
{
	if(input[i]==(1105+offset2))
{
	vR[i] = 1105;
}
else
{
	vR[i] = 1104;
}

}

}

}

}
else
{
	if(input[i]>(1095+offset2))
{
	if(input[i]>(1099+offset2))
{
	if(input[i]>(1101+offset2))
{
	if(input[i]==(1103+offset2))
{
	vR[i] = 1103;
}
else
{
	vR[i] = 1102;
}

}
else
{
	if(input[i]==(1101+offset2))
{
	vR[i] = 1101;
}
else
{
	vR[i] = 1100;
}

}

}
else
{
	if(input[i]>(1097+offset2))
{
	if(input[i]==(1099+offset2))
{
	vR[i] = 1099;
}
else
{
	vR[i] = 1098;
}

}
else
{
	if(input[i]==(1097+offset2))
{
	vR[i] = 1097;
}
else
{
	vR[i] = 1096;
}

}

}

}
else
{
	if(input[i]>(1091+offset2))
{
	if(input[i]>(1093+offset2))
{
	if(input[i]==(1095+offset2))
{
	vR[i] = 1095;
}
else
{
	vR[i] = 1094;
}

}
else
{
	if(input[i]==(1093+offset2))
{
	vR[i] = 1093;
}
else
{
	vR[i] = 1092;
}

}

}
else
{
	if(input[i]>(1089+offset2))
{
	if(input[i]==(1091+offset2))
{
	vR[i] = 1091;
}
else
{
	vR[i] = 1090;
}

}
else
{
	if(input[i]==(1089+offset2))
{
	vR[i] = 1089;
}
else
{
	vR[i] = 1088;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1055+offset2))
{
	if(input[i]>(1071+offset2))
{
	if(input[i]>(1079+offset2))
{
	if(input[i]>(1083+offset2))
{
	if(input[i]>(1085+offset2))
{
	if(input[i]==(1087+offset2))
{
	vR[i] = 1087;
}
else
{
	vR[i] = 1086;
}

}
else
{
	if(input[i]==(1085+offset2))
{
	vR[i] = 1085;
}
else
{
	vR[i] = 1084;
}

}

}
else
{
	if(input[i]>(1081+offset2))
{
	if(input[i]==(1083+offset2))
{
	vR[i] = 1083;
}
else
{
	vR[i] = 1082;
}

}
else
{
	if(input[i]==(1081+offset2))
{
	vR[i] = 1081;
}
else
{
	vR[i] = 1080;
}

}

}

}
else
{
	if(input[i]>(1075+offset2))
{
	if(input[i]>(1077+offset2))
{
	if(input[i]==(1079+offset2))
{
	vR[i] = 1079;
}
else
{
	vR[i] = 1078;
}

}
else
{
	if(input[i]==(1077+offset2))
{
	vR[i] = 1077;
}
else
{
	vR[i] = 1076;
}

}

}
else
{
	if(input[i]>(1073+offset2))
{
	if(input[i]==(1075+offset2))
{
	vR[i] = 1075;
}
else
{
	vR[i] = 1074;
}

}
else
{
	if(input[i]==(1073+offset2))
{
	vR[i] = 1073;
}
else
{
	vR[i] = 1072;
}

}

}

}

}
else
{
	if(input[i]>(1063+offset2))
{
	if(input[i]>(1067+offset2))
{
	if(input[i]>(1069+offset2))
{
	if(input[i]==(1071+offset2))
{
	vR[i] = 1071;
}
else
{
	vR[i] = 1070;
}

}
else
{
	if(input[i]==(1069+offset2))
{
	vR[i] = 1069;
}
else
{
	vR[i] = 1068;
}

}

}
else
{
	if(input[i]>(1065+offset2))
{
	if(input[i]==(1067+offset2))
{
	vR[i] = 1067;
}
else
{
	vR[i] = 1066;
}

}
else
{
	if(input[i]==(1065+offset2))
{
	vR[i] = 1065;
}
else
{
	vR[i] = 1064;
}

}

}

}
else
{
	if(input[i]>(1059+offset2))
{
	if(input[i]>(1061+offset2))
{
	if(input[i]==(1063+offset2))
{
	vR[i] = 1063;
}
else
{
	vR[i] = 1062;
}

}
else
{
	if(input[i]==(1061+offset2))
{
	vR[i] = 1061;
}
else
{
	vR[i] = 1060;
}

}

}
else
{
	if(input[i]>(1057+offset2))
{
	if(input[i]==(1059+offset2))
{
	vR[i] = 1059;
}
else
{
	vR[i] = 1058;
}

}
else
{
	if(input[i]==(1057+offset2))
{
	vR[i] = 1057;
}
else
{
	vR[i] = 1056;
}

}

}

}

}

}
else
{
	if(input[i]>(1039+offset2))
{
	if(input[i]>(1047+offset2))
{
	if(input[i]>(1051+offset2))
{
	if(input[i]>(1053+offset2))
{
	if(input[i]==(1055+offset2))
{
	vR[i] = 1055;
}
else
{
	vR[i] = 1054;
}

}
else
{
	if(input[i]==(1053+offset2))
{
	vR[i] = 1053;
}
else
{
	vR[i] = 1052;
}

}

}
else
{
	if(input[i]>(1049+offset2))
{
	if(input[i]==(1051+offset2))
{
	vR[i] = 1051;
}
else
{
	vR[i] = 1050;
}

}
else
{
	if(input[i]==(1049+offset2))
{
	vR[i] = 1049;
}
else
{
	vR[i] = 1048;
}

}

}

}
else
{
	if(input[i]>(1043+offset2))
{
	if(input[i]>(1045+offset2))
{
	if(input[i]==(1047+offset2))
{
	vR[i] = 1047;
}
else
{
	vR[i] = 1046;
}

}
else
{
	if(input[i]==(1045+offset2))
{
	vR[i] = 1045;
}
else
{
	vR[i] = 1044;
}

}

}
else
{
	if(input[i]>(1041+offset2))
{
	if(input[i]==(1043+offset2))
{
	vR[i] = 1043;
}
else
{
	vR[i] = 1042;
}

}
else
{
	if(input[i]==(1041+offset2))
{
	vR[i] = 1041;
}
else
{
	vR[i] = 1040;
}

}

}

}

}
else
{
	if(input[i]>(1031+offset2))
{
	if(input[i]>(1035+offset2))
{
	if(input[i]>(1037+offset2))
{
	if(input[i]==(1039+offset2))
{
	vR[i] = 1039;
}
else
{
	vR[i] = 1038;
}

}
else
{
	if(input[i]==(1037+offset2))
{
	vR[i] = 1037;
}
else
{
	vR[i] = 1036;
}

}

}
else
{
	if(input[i]>(1033+offset2))
{
	if(input[i]==(1035+offset2))
{
	vR[i] = 1035;
}
else
{
	vR[i] = 1034;
}

}
else
{
	if(input[i]==(1033+offset2))
{
	vR[i] = 1033;
}
else
{
	vR[i] = 1032;
}

}

}

}
else
{
	if(input[i]>(1027+offset2))
{
	if(input[i]>(1029+offset2))
{
	if(input[i]==(1031+offset2))
{
	vR[i] = 1031;
}
else
{
	vR[i] = 1030;
}

}
else
{
	if(input[i]==(1029+offset2))
{
	vR[i] = 1029;
}
else
{
	vR[i] = 1028;
}

}

}
else
{
	if(input[i]>(1025+offset2))
{
	if(input[i]==(1027+offset2))
{
	vR[i] = 1027;
}
else
{
	vR[i] = 1026;
}

}
else
{
	if(input[i]==(1025+offset2))
{
	vR[i] = 1025;
}
else
{
	vR[i] = 1024;
}

}

}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(511+offset2))
{
	if(input[i]>(767+offset2))
{
	if(input[i]>(895+offset2))
{
	if(input[i]>(959+offset2))
{
	if(input[i]>(991+offset2))
{
	if(input[i]>(1007+offset2))
{
	if(input[i]>(1015+offset2))
{
	if(input[i]>(1019+offset2))
{
	if(input[i]>(1021+offset2))
{
	if(input[i]==(1023+offset2))
{
	vR[i] = 1023;
}
else
{
	vR[i] = 1022;
}

}
else
{
	if(input[i]==(1021+offset2))
{
	vR[i] = 1021;
}
else
{
	vR[i] = 1020;
}

}

}
else
{
	if(input[i]>(1017+offset2))
{
	if(input[i]==(1019+offset2))
{
	vR[i] = 1019;
}
else
{
	vR[i] = 1018;
}

}
else
{
	if(input[i]==(1017+offset2))
{
	vR[i] = 1017;
}
else
{
	vR[i] = 1016;
}

}

}

}
else
{
	if(input[i]>(1011+offset2))
{
	if(input[i]>(1013+offset2))
{
	if(input[i]==(1015+offset2))
{
	vR[i] = 1015;
}
else
{
	vR[i] = 1014;
}

}
else
{
	if(input[i]==(1013+offset2))
{
	vR[i] = 1013;
}
else
{
	vR[i] = 1012;
}

}

}
else
{
	if(input[i]>(1009+offset2))
{
	if(input[i]==(1011+offset2))
{
	vR[i] = 1011;
}
else
{
	vR[i] = 1010;
}

}
else
{
	if(input[i]==(1009+offset2))
{
	vR[i] = 1009;
}
else
{
	vR[i] = 1008;
}

}

}

}

}
else
{
	if(input[i]>(999+offset2))
{
	if(input[i]>(1003+offset2))
{
	if(input[i]>(1005+offset2))
{
	if(input[i]==(1007+offset2))
{
	vR[i] = 1007;
}
else
{
	vR[i] = 1006;
}

}
else
{
	if(input[i]==(1005+offset2))
{
	vR[i] = 1005;
}
else
{
	vR[i] = 1004;
}

}

}
else
{
	if(input[i]>(1001+offset2))
{
	if(input[i]==(1003+offset2))
{
	vR[i] = 1003;
}
else
{
	vR[i] = 1002;
}

}
else
{
	if(input[i]==(1001+offset2))
{
	vR[i] = 1001;
}
else
{
	vR[i] = 1000;
}

}

}

}
else
{
	if(input[i]>(995+offset2))
{
	if(input[i]>(997+offset2))
{
	if(input[i]==(999+offset2))
{
	vR[i] = 999;
}
else
{
	vR[i] = 998;
}

}
else
{
	if(input[i]==(997+offset2))
{
	vR[i] = 997;
}
else
{
	vR[i] = 996;
}

}

}
else
{
	if(input[i]>(993+offset2))
{
	if(input[i]==(995+offset2))
{
	vR[i] = 995;
}
else
{
	vR[i] = 994;
}

}
else
{
	if(input[i]==(993+offset2))
{
	vR[i] = 993;
}
else
{
	vR[i] = 992;
}

}

}

}

}

}
else
{
	if(input[i]>(975+offset2))
{
	if(input[i]>(983+offset2))
{
	if(input[i]>(987+offset2))
{
	if(input[i]>(989+offset2))
{
	if(input[i]==(991+offset2))
{
	vR[i] = 991;
}
else
{
	vR[i] = 990;
}

}
else
{
	if(input[i]==(989+offset2))
{
	vR[i] = 989;
}
else
{
	vR[i] = 988;
}

}

}
else
{
	if(input[i]>(985+offset2))
{
	if(input[i]==(987+offset2))
{
	vR[i] = 987;
}
else
{
	vR[i] = 986;
}

}
else
{
	if(input[i]==(985+offset2))
{
	vR[i] = 985;
}
else
{
	vR[i] = 984;
}

}

}

}
else
{
	if(input[i]>(979+offset2))
{
	if(input[i]>(981+offset2))
{
	if(input[i]==(983+offset2))
{
	vR[i] = 983;
}
else
{
	vR[i] = 982;
}

}
else
{
	if(input[i]==(981+offset2))
{
	vR[i] = 981;
}
else
{
	vR[i] = 980;
}

}

}
else
{
	if(input[i]>(977+offset2))
{
	if(input[i]==(979+offset2))
{
	vR[i] = 979;
}
else
{
	vR[i] = 978;
}

}
else
{
	if(input[i]==(977+offset2))
{
	vR[i] = 977;
}
else
{
	vR[i] = 976;
}

}

}

}

}
else
{
	if(input[i]>(967+offset2))
{
	if(input[i]>(971+offset2))
{
	if(input[i]>(973+offset2))
{
	if(input[i]==(975+offset2))
{
	vR[i] = 975;
}
else
{
	vR[i] = 974;
}

}
else
{
	if(input[i]==(973+offset2))
{
	vR[i] = 973;
}
else
{
	vR[i] = 972;
}

}

}
else
{
	if(input[i]>(969+offset2))
{
	if(input[i]==(971+offset2))
{
	vR[i] = 971;
}
else
{
	vR[i] = 970;
}

}
else
{
	if(input[i]==(969+offset2))
{
	vR[i] = 969;
}
else
{
	vR[i] = 968;
}

}

}

}
else
{
	if(input[i]>(963+offset2))
{
	if(input[i]>(965+offset2))
{
	if(input[i]==(967+offset2))
{
	vR[i] = 967;
}
else
{
	vR[i] = 966;
}

}
else
{
	if(input[i]==(965+offset2))
{
	vR[i] = 965;
}
else
{
	vR[i] = 964;
}

}

}
else
{
	if(input[i]>(961+offset2))
{
	if(input[i]==(963+offset2))
{
	vR[i] = 963;
}
else
{
	vR[i] = 962;
}

}
else
{
	if(input[i]==(961+offset2))
{
	vR[i] = 961;
}
else
{
	vR[i] = 960;
}

}

}

}

}

}

}
else
{
	if(input[i]>(927+offset2))
{
	if(input[i]>(943+offset2))
{
	if(input[i]>(951+offset2))
{
	if(input[i]>(955+offset2))
{
	if(input[i]>(957+offset2))
{
	if(input[i]==(959+offset2))
{
	vR[i] = 959;
}
else
{
	vR[i] = 958;
}

}
else
{
	if(input[i]==(957+offset2))
{
	vR[i] = 957;
}
else
{
	vR[i] = 956;
}

}

}
else
{
	if(input[i]>(953+offset2))
{
	if(input[i]==(955+offset2))
{
	vR[i] = 955;
}
else
{
	vR[i] = 954;
}

}
else
{
	if(input[i]==(953+offset2))
{
	vR[i] = 953;
}
else
{
	vR[i] = 952;
}

}

}

}
else
{
	if(input[i]>(947+offset2))
{
	if(input[i]>(949+offset2))
{
	if(input[i]==(951+offset2))
{
	vR[i] = 951;
}
else
{
	vR[i] = 950;
}

}
else
{
	if(input[i]==(949+offset2))
{
	vR[i] = 949;
}
else
{
	vR[i] = 948;
}

}

}
else
{
	if(input[i]>(945+offset2))
{
	if(input[i]==(947+offset2))
{
	vR[i] = 947;
}
else
{
	vR[i] = 946;
}

}
else
{
	if(input[i]==(945+offset2))
{
	vR[i] = 945;
}
else
{
	vR[i] = 944;
}

}

}

}

}
else
{
	if(input[i]>(935+offset2))
{
	if(input[i]>(939+offset2))
{
	if(input[i]>(941+offset2))
{
	if(input[i]==(943+offset2))
{
	vR[i] = 943;
}
else
{
	vR[i] = 942;
}

}
else
{
	if(input[i]==(941+offset2))
{
	vR[i] = 941;
}
else
{
	vR[i] = 940;
}

}

}
else
{
	if(input[i]>(937+offset2))
{
	if(input[i]==(939+offset2))
{
	vR[i] = 939;
}
else
{
	vR[i] = 938;
}

}
else
{
	if(input[i]==(937+offset2))
{
	vR[i] = 937;
}
else
{
	vR[i] = 936;
}

}

}

}
else
{
	if(input[i]>(931+offset2))
{
	if(input[i]>(933+offset2))
{
	if(input[i]==(935+offset2))
{
	vR[i] = 935;
}
else
{
	vR[i] = 934;
}

}
else
{
	if(input[i]==(933+offset2))
{
	vR[i] = 933;
}
else
{
	vR[i] = 932;
}

}

}
else
{
	if(input[i]>(929+offset2))
{
	if(input[i]==(931+offset2))
{
	vR[i] = 931;
}
else
{
	vR[i] = 930;
}

}
else
{
	if(input[i]==(929+offset2))
{
	vR[i] = 929;
}
else
{
	vR[i] = 928;
}

}

}

}

}

}
else
{
	if(input[i]>(911+offset2))
{
	if(input[i]>(919+offset2))
{
	if(input[i]>(923+offset2))
{
	if(input[i]>(925+offset2))
{
	if(input[i]==(927+offset2))
{
	vR[i] = 927;
}
else
{
	vR[i] = 926;
}

}
else
{
	if(input[i]==(925+offset2))
{
	vR[i] = 925;
}
else
{
	vR[i] = 924;
}

}

}
else
{
	if(input[i]>(921+offset2))
{
	if(input[i]==(923+offset2))
{
	vR[i] = 923;
}
else
{
	vR[i] = 922;
}

}
else
{
	if(input[i]==(921+offset2))
{
	vR[i] = 921;
}
else
{
	vR[i] = 920;
}

}

}

}
else
{
	if(input[i]>(915+offset2))
{
	if(input[i]>(917+offset2))
{
	if(input[i]==(919+offset2))
{
	vR[i] = 919;
}
else
{
	vR[i] = 918;
}

}
else
{
	if(input[i]==(917+offset2))
{
	vR[i] = 917;
}
else
{
	vR[i] = 916;
}

}

}
else
{
	if(input[i]>(913+offset2))
{
	if(input[i]==(915+offset2))
{
	vR[i] = 915;
}
else
{
	vR[i] = 914;
}

}
else
{
	if(input[i]==(913+offset2))
{
	vR[i] = 913;
}
else
{
	vR[i] = 912;
}

}

}

}

}
else
{
	if(input[i]>(903+offset2))
{
	if(input[i]>(907+offset2))
{
	if(input[i]>(909+offset2))
{
	if(input[i]==(911+offset2))
{
	vR[i] = 911;
}
else
{
	vR[i] = 910;
}

}
else
{
	if(input[i]==(909+offset2))
{
	vR[i] = 909;
}
else
{
	vR[i] = 908;
}

}

}
else
{
	if(input[i]>(905+offset2))
{
	if(input[i]==(907+offset2))
{
	vR[i] = 907;
}
else
{
	vR[i] = 906;
}

}
else
{
	if(input[i]==(905+offset2))
{
	vR[i] = 905;
}
else
{
	vR[i] = 904;
}

}

}

}
else
{
	if(input[i]>(899+offset2))
{
	if(input[i]>(901+offset2))
{
	if(input[i]==(903+offset2))
{
	vR[i] = 903;
}
else
{
	vR[i] = 902;
}

}
else
{
	if(input[i]==(901+offset2))
{
	vR[i] = 901;
}
else
{
	vR[i] = 900;
}

}

}
else
{
	if(input[i]>(897+offset2))
{
	if(input[i]==(899+offset2))
{
	vR[i] = 899;
}
else
{
	vR[i] = 898;
}

}
else
{
	if(input[i]==(897+offset2))
{
	vR[i] = 897;
}
else
{
	vR[i] = 896;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(831+offset2))
{
	if(input[i]>(863+offset2))
{
	if(input[i]>(879+offset2))
{
	if(input[i]>(887+offset2))
{
	if(input[i]>(891+offset2))
{
	if(input[i]>(893+offset2))
{
	if(input[i]==(895+offset2))
{
	vR[i] = 895;
}
else
{
	vR[i] = 894;
}

}
else
{
	if(input[i]==(893+offset2))
{
	vR[i] = 893;
}
else
{
	vR[i] = 892;
}

}

}
else
{
	if(input[i]>(889+offset2))
{
	if(input[i]==(891+offset2))
{
	vR[i] = 891;
}
else
{
	vR[i] = 890;
}

}
else
{
	if(input[i]==(889+offset2))
{
	vR[i] = 889;
}
else
{
	vR[i] = 888;
}

}

}

}
else
{
	if(input[i]>(883+offset2))
{
	if(input[i]>(885+offset2))
{
	if(input[i]==(887+offset2))
{
	vR[i] = 887;
}
else
{
	vR[i] = 886;
}

}
else
{
	if(input[i]==(885+offset2))
{
	vR[i] = 885;
}
else
{
	vR[i] = 884;
}

}

}
else
{
	if(input[i]>(881+offset2))
{
	if(input[i]==(883+offset2))
{
	vR[i] = 883;
}
else
{
	vR[i] = 882;
}

}
else
{
	if(input[i]==(881+offset2))
{
	vR[i] = 881;
}
else
{
	vR[i] = 880;
}

}

}

}

}
else
{
	if(input[i]>(871+offset2))
{
	if(input[i]>(875+offset2))
{
	if(input[i]>(877+offset2))
{
	if(input[i]==(879+offset2))
{
	vR[i] = 879;
}
else
{
	vR[i] = 878;
}

}
else
{
	if(input[i]==(877+offset2))
{
	vR[i] = 877;
}
else
{
	vR[i] = 876;
}

}

}
else
{
	if(input[i]>(873+offset2))
{
	if(input[i]==(875+offset2))
{
	vR[i] = 875;
}
else
{
	vR[i] = 874;
}

}
else
{
	if(input[i]==(873+offset2))
{
	vR[i] = 873;
}
else
{
	vR[i] = 872;
}

}

}

}
else
{
	if(input[i]>(867+offset2))
{
	if(input[i]>(869+offset2))
{
	if(input[i]==(871+offset2))
{
	vR[i] = 871;
}
else
{
	vR[i] = 870;
}

}
else
{
	if(input[i]==(869+offset2))
{
	vR[i] = 869;
}
else
{
	vR[i] = 868;
}

}

}
else
{
	if(input[i]>(865+offset2))
{
	if(input[i]==(867+offset2))
{
	vR[i] = 867;
}
else
{
	vR[i] = 866;
}

}
else
{
	if(input[i]==(865+offset2))
{
	vR[i] = 865;
}
else
{
	vR[i] = 864;
}

}

}

}

}

}
else
{
	if(input[i]>(847+offset2))
{
	if(input[i]>(855+offset2))
{
	if(input[i]>(859+offset2))
{
	if(input[i]>(861+offset2))
{
	if(input[i]==(863+offset2))
{
	vR[i] = 863;
}
else
{
	vR[i] = 862;
}

}
else
{
	if(input[i]==(861+offset2))
{
	vR[i] = 861;
}
else
{
	vR[i] = 860;
}

}

}
else
{
	if(input[i]>(857+offset2))
{
	if(input[i]==(859+offset2))
{
	vR[i] = 859;
}
else
{
	vR[i] = 858;
}

}
else
{
	if(input[i]==(857+offset2))
{
	vR[i] = 857;
}
else
{
	vR[i] = 856;
}

}

}

}
else
{
	if(input[i]>(851+offset2))
{
	if(input[i]>(853+offset2))
{
	if(input[i]==(855+offset2))
{
	vR[i] = 855;
}
else
{
	vR[i] = 854;
}

}
else
{
	if(input[i]==(853+offset2))
{
	vR[i] = 853;
}
else
{
	vR[i] = 852;
}

}

}
else
{
	if(input[i]>(849+offset2))
{
	if(input[i]==(851+offset2))
{
	vR[i] = 851;
}
else
{
	vR[i] = 850;
}

}
else
{
	if(input[i]==(849+offset2))
{
	vR[i] = 849;
}
else
{
	vR[i] = 848;
}

}

}

}

}
else
{
	if(input[i]>(839+offset2))
{
	if(input[i]>(843+offset2))
{
	if(input[i]>(845+offset2))
{
	if(input[i]==(847+offset2))
{
	vR[i] = 847;
}
else
{
	vR[i] = 846;
}

}
else
{
	if(input[i]==(845+offset2))
{
	vR[i] = 845;
}
else
{
	vR[i] = 844;
}

}

}
else
{
	if(input[i]>(841+offset2))
{
	if(input[i]==(843+offset2))
{
	vR[i] = 843;
}
else
{
	vR[i] = 842;
}

}
else
{
	if(input[i]==(841+offset2))
{
	vR[i] = 841;
}
else
{
	vR[i] = 840;
}

}

}

}
else
{
	if(input[i]>(835+offset2))
{
	if(input[i]>(837+offset2))
{
	if(input[i]==(839+offset2))
{
	vR[i] = 839;
}
else
{
	vR[i] = 838;
}

}
else
{
	if(input[i]==(837+offset2))
{
	vR[i] = 837;
}
else
{
	vR[i] = 836;
}

}

}
else
{
	if(input[i]>(833+offset2))
{
	if(input[i]==(835+offset2))
{
	vR[i] = 835;
}
else
{
	vR[i] = 834;
}

}
else
{
	if(input[i]==(833+offset2))
{
	vR[i] = 833;
}
else
{
	vR[i] = 832;
}

}

}

}

}

}

}
else
{
	if(input[i]>(799+offset2))
{
	if(input[i]>(815+offset2))
{
	if(input[i]>(823+offset2))
{
	if(input[i]>(827+offset2))
{
	if(input[i]>(829+offset2))
{
	if(input[i]==(831+offset2))
{
	vR[i] = 831;
}
else
{
	vR[i] = 830;
}

}
else
{
	if(input[i]==(829+offset2))
{
	vR[i] = 829;
}
else
{
	vR[i] = 828;
}

}

}
else
{
	if(input[i]>(825+offset2))
{
	if(input[i]==(827+offset2))
{
	vR[i] = 827;
}
else
{
	vR[i] = 826;
}

}
else
{
	if(input[i]==(825+offset2))
{
	vR[i] = 825;
}
else
{
	vR[i] = 824;
}

}

}

}
else
{
	if(input[i]>(819+offset2))
{
	if(input[i]>(821+offset2))
{
	if(input[i]==(823+offset2))
{
	vR[i] = 823;
}
else
{
	vR[i] = 822;
}

}
else
{
	if(input[i]==(821+offset2))
{
	vR[i] = 821;
}
else
{
	vR[i] = 820;
}

}

}
else
{
	if(input[i]>(817+offset2))
{
	if(input[i]==(819+offset2))
{
	vR[i] = 819;
}
else
{
	vR[i] = 818;
}

}
else
{
	if(input[i]==(817+offset2))
{
	vR[i] = 817;
}
else
{
	vR[i] = 816;
}

}

}

}

}
else
{
	if(input[i]>(807+offset2))
{
	if(input[i]>(811+offset2))
{
	if(input[i]>(813+offset2))
{
	if(input[i]==(815+offset2))
{
	vR[i] = 815;
}
else
{
	vR[i] = 814;
}

}
else
{
	if(input[i]==(813+offset2))
{
	vR[i] = 813;
}
else
{
	vR[i] = 812;
}

}

}
else
{
	if(input[i]>(809+offset2))
{
	if(input[i]==(811+offset2))
{
	vR[i] = 811;
}
else
{
	vR[i] = 810;
}

}
else
{
	if(input[i]==(809+offset2))
{
	vR[i] = 809;
}
else
{
	vR[i] = 808;
}

}

}

}
else
{
	if(input[i]>(803+offset2))
{
	if(input[i]>(805+offset2))
{
	if(input[i]==(807+offset2))
{
	vR[i] = 807;
}
else
{
	vR[i] = 806;
}

}
else
{
	if(input[i]==(805+offset2))
{
	vR[i] = 805;
}
else
{
	vR[i] = 804;
}

}

}
else
{
	if(input[i]>(801+offset2))
{
	if(input[i]==(803+offset2))
{
	vR[i] = 803;
}
else
{
	vR[i] = 802;
}

}
else
{
	if(input[i]==(801+offset2))
{
	vR[i] = 801;
}
else
{
	vR[i] = 800;
}

}

}

}

}

}
else
{
	if(input[i]>(783+offset2))
{
	if(input[i]>(791+offset2))
{
	if(input[i]>(795+offset2))
{
	if(input[i]>(797+offset2))
{
	if(input[i]==(799+offset2))
{
	vR[i] = 799;
}
else
{
	vR[i] = 798;
}

}
else
{
	if(input[i]==(797+offset2))
{
	vR[i] = 797;
}
else
{
	vR[i] = 796;
}

}

}
else
{
	if(input[i]>(793+offset2))
{
	if(input[i]==(795+offset2))
{
	vR[i] = 795;
}
else
{
	vR[i] = 794;
}

}
else
{
	if(input[i]==(793+offset2))
{
	vR[i] = 793;
}
else
{
	vR[i] = 792;
}

}

}

}
else
{
	if(input[i]>(787+offset2))
{
	if(input[i]>(789+offset2))
{
	if(input[i]==(791+offset2))
{
	vR[i] = 791;
}
else
{
	vR[i] = 790;
}

}
else
{
	if(input[i]==(789+offset2))
{
	vR[i] = 789;
}
else
{
	vR[i] = 788;
}

}

}
else
{
	if(input[i]>(785+offset2))
{
	if(input[i]==(787+offset2))
{
	vR[i] = 787;
}
else
{
	vR[i] = 786;
}

}
else
{
	if(input[i]==(785+offset2))
{
	vR[i] = 785;
}
else
{
	vR[i] = 784;
}

}

}

}

}
else
{
	if(input[i]>(775+offset2))
{
	if(input[i]>(779+offset2))
{
	if(input[i]>(781+offset2))
{
	if(input[i]==(783+offset2))
{
	vR[i] = 783;
}
else
{
	vR[i] = 782;
}

}
else
{
	if(input[i]==(781+offset2))
{
	vR[i] = 781;
}
else
{
	vR[i] = 780;
}

}

}
else
{
	if(input[i]>(777+offset2))
{
	if(input[i]==(779+offset2))
{
	vR[i] = 779;
}
else
{
	vR[i] = 778;
}

}
else
{
	if(input[i]==(777+offset2))
{
	vR[i] = 777;
}
else
{
	vR[i] = 776;
}

}

}

}
else
{
	if(input[i]>(771+offset2))
{
	if(input[i]>(773+offset2))
{
	if(input[i]==(775+offset2))
{
	vR[i] = 775;
}
else
{
	vR[i] = 774;
}

}
else
{
	if(input[i]==(773+offset2))
{
	vR[i] = 773;
}
else
{
	vR[i] = 772;
}

}

}
else
{
	if(input[i]>(769+offset2))
{
	if(input[i]==(771+offset2))
{
	vR[i] = 771;
}
else
{
	vR[i] = 770;
}

}
else
{
	if(input[i]==(769+offset2))
{
	vR[i] = 769;
}
else
{
	vR[i] = 768;
}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(639+offset2))
{
	if(input[i]>(703+offset2))
{
	if(input[i]>(735+offset2))
{
	if(input[i]>(751+offset2))
{
	if(input[i]>(759+offset2))
{
	if(input[i]>(763+offset2))
{
	if(input[i]>(765+offset2))
{
	if(input[i]==(767+offset2))
{
	vR[i] = 767;
}
else
{
	vR[i] = 766;
}

}
else
{
	if(input[i]==(765+offset2))
{
	vR[i] = 765;
}
else
{
	vR[i] = 764;
}

}

}
else
{
	if(input[i]>(761+offset2))
{
	if(input[i]==(763+offset2))
{
	vR[i] = 763;
}
else
{
	vR[i] = 762;
}

}
else
{
	if(input[i]==(761+offset2))
{
	vR[i] = 761;
}
else
{
	vR[i] = 760;
}

}

}

}
else
{
	if(input[i]>(755+offset2))
{
	if(input[i]>(757+offset2))
{
	if(input[i]==(759+offset2))
{
	vR[i] = 759;
}
else
{
	vR[i] = 758;
}

}
else
{
	if(input[i]==(757+offset2))
{
	vR[i] = 757;
}
else
{
	vR[i] = 756;
}

}

}
else
{
	if(input[i]>(753+offset2))
{
	if(input[i]==(755+offset2))
{
	vR[i] = 755;
}
else
{
	vR[i] = 754;
}

}
else
{
	if(input[i]==(753+offset2))
{
	vR[i] = 753;
}
else
{
	vR[i] = 752;
}

}

}

}

}
else
{
	if(input[i]>(743+offset2))
{
	if(input[i]>(747+offset2))
{
	if(input[i]>(749+offset2))
{
	if(input[i]==(751+offset2))
{
	vR[i] = 751;
}
else
{
	vR[i] = 750;
}

}
else
{
	if(input[i]==(749+offset2))
{
	vR[i] = 749;
}
else
{
	vR[i] = 748;
}

}

}
else
{
	if(input[i]>(745+offset2))
{
	if(input[i]==(747+offset2))
{
	vR[i] = 747;
}
else
{
	vR[i] = 746;
}

}
else
{
	if(input[i]==(745+offset2))
{
	vR[i] = 745;
}
else
{
	vR[i] = 744;
}

}

}

}
else
{
	if(input[i]>(739+offset2))
{
	if(input[i]>(741+offset2))
{
	if(input[i]==(743+offset2))
{
	vR[i] = 743;
}
else
{
	vR[i] = 742;
}

}
else
{
	if(input[i]==(741+offset2))
{
	vR[i] = 741;
}
else
{
	vR[i] = 740;
}

}

}
else
{
	if(input[i]>(737+offset2))
{
	if(input[i]==(739+offset2))
{
	vR[i] = 739;
}
else
{
	vR[i] = 738;
}

}
else
{
	if(input[i]==(737+offset2))
{
	vR[i] = 737;
}
else
{
	vR[i] = 736;
}

}

}

}

}

}
else
{
	if(input[i]>(719+offset2))
{
	if(input[i]>(727+offset2))
{
	if(input[i]>(731+offset2))
{
	if(input[i]>(733+offset2))
{
	if(input[i]==(735+offset2))
{
	vR[i] = 735;
}
else
{
	vR[i] = 734;
}

}
else
{
	if(input[i]==(733+offset2))
{
	vR[i] = 733;
}
else
{
	vR[i] = 732;
}

}

}
else
{
	if(input[i]>(729+offset2))
{
	if(input[i]==(731+offset2))
{
	vR[i] = 731;
}
else
{
	vR[i] = 730;
}

}
else
{
	if(input[i]==(729+offset2))
{
	vR[i] = 729;
}
else
{
	vR[i] = 728;
}

}

}

}
else
{
	if(input[i]>(723+offset2))
{
	if(input[i]>(725+offset2))
{
	if(input[i]==(727+offset2))
{
	vR[i] = 727;
}
else
{
	vR[i] = 726;
}

}
else
{
	if(input[i]==(725+offset2))
{
	vR[i] = 725;
}
else
{
	vR[i] = 724;
}

}

}
else
{
	if(input[i]>(721+offset2))
{
	if(input[i]==(723+offset2))
{
	vR[i] = 723;
}
else
{
	vR[i] = 722;
}

}
else
{
	if(input[i]==(721+offset2))
{
	vR[i] = 721;
}
else
{
	vR[i] = 720;
}

}

}

}

}
else
{
	if(input[i]>(711+offset2))
{
	if(input[i]>(715+offset2))
{
	if(input[i]>(717+offset2))
{
	if(input[i]==(719+offset2))
{
	vR[i] = 719;
}
else
{
	vR[i] = 718;
}

}
else
{
	if(input[i]==(717+offset2))
{
	vR[i] = 717;
}
else
{
	vR[i] = 716;
}

}

}
else
{
	if(input[i]>(713+offset2))
{
	if(input[i]==(715+offset2))
{
	vR[i] = 715;
}
else
{
	vR[i] = 714;
}

}
else
{
	if(input[i]==(713+offset2))
{
	vR[i] = 713;
}
else
{
	vR[i] = 712;
}

}

}

}
else
{
	if(input[i]>(707+offset2))
{
	if(input[i]>(709+offset2))
{
	if(input[i]==(711+offset2))
{
	vR[i] = 711;
}
else
{
	vR[i] = 710;
}

}
else
{
	if(input[i]==(709+offset2))
{
	vR[i] = 709;
}
else
{
	vR[i] = 708;
}

}

}
else
{
	if(input[i]>(705+offset2))
{
	if(input[i]==(707+offset2))
{
	vR[i] = 707;
}
else
{
	vR[i] = 706;
}

}
else
{
	if(input[i]==(705+offset2))
{
	vR[i] = 705;
}
else
{
	vR[i] = 704;
}

}

}

}

}

}

}
else
{
	if(input[i]>(671+offset2))
{
	if(input[i]>(687+offset2))
{
	if(input[i]>(695+offset2))
{
	if(input[i]>(699+offset2))
{
	if(input[i]>(701+offset2))
{
	if(input[i]==(703+offset2))
{
	vR[i] = 703;
}
else
{
	vR[i] = 702;
}

}
else
{
	if(input[i]==(701+offset2))
{
	vR[i] = 701;
}
else
{
	vR[i] = 700;
}

}

}
else
{
	if(input[i]>(697+offset2))
{
	if(input[i]==(699+offset2))
{
	vR[i] = 699;
}
else
{
	vR[i] = 698;
}

}
else
{
	if(input[i]==(697+offset2))
{
	vR[i] = 697;
}
else
{
	vR[i] = 696;
}

}

}

}
else
{
	if(input[i]>(691+offset2))
{
	if(input[i]>(693+offset2))
{
	if(input[i]==(695+offset2))
{
	vR[i] = 695;
}
else
{
	vR[i] = 694;
}

}
else
{
	if(input[i]==(693+offset2))
{
	vR[i] = 693;
}
else
{
	vR[i] = 692;
}

}

}
else
{
	if(input[i]>(689+offset2))
{
	if(input[i]==(691+offset2))
{
	vR[i] = 691;
}
else
{
	vR[i] = 690;
}

}
else
{
	if(input[i]==(689+offset2))
{
	vR[i] = 689;
}
else
{
	vR[i] = 688;
}

}

}

}

}
else
{
	if(input[i]>(679+offset2))
{
	if(input[i]>(683+offset2))
{
	if(input[i]>(685+offset2))
{
	if(input[i]==(687+offset2))
{
	vR[i] = 687;
}
else
{
	vR[i] = 686;
}

}
else
{
	if(input[i]==(685+offset2))
{
	vR[i] = 685;
}
else
{
	vR[i] = 684;
}

}

}
else
{
	if(input[i]>(681+offset2))
{
	if(input[i]==(683+offset2))
{
	vR[i] = 683;
}
else
{
	vR[i] = 682;
}

}
else
{
	if(input[i]==(681+offset2))
{
	vR[i] = 681;
}
else
{
	vR[i] = 680;
}

}

}

}
else
{
	if(input[i]>(675+offset2))
{
	if(input[i]>(677+offset2))
{
	if(input[i]==(679+offset2))
{
	vR[i] = 679;
}
else
{
	vR[i] = 678;
}

}
else
{
	if(input[i]==(677+offset2))
{
	vR[i] = 677;
}
else
{
	vR[i] = 676;
}

}

}
else
{
	if(input[i]>(673+offset2))
{
	if(input[i]==(675+offset2))
{
	vR[i] = 675;
}
else
{
	vR[i] = 674;
}

}
else
{
	if(input[i]==(673+offset2))
{
	vR[i] = 673;
}
else
{
	vR[i] = 672;
}

}

}

}

}

}
else
{
	if(input[i]>(655+offset2))
{
	if(input[i]>(663+offset2))
{
	if(input[i]>(667+offset2))
{
	if(input[i]>(669+offset2))
{
	if(input[i]==(671+offset2))
{
	vR[i] = 671;
}
else
{
	vR[i] = 670;
}

}
else
{
	if(input[i]==(669+offset2))
{
	vR[i] = 669;
}
else
{
	vR[i] = 668;
}

}

}
else
{
	if(input[i]>(665+offset2))
{
	if(input[i]==(667+offset2))
{
	vR[i] = 667;
}
else
{
	vR[i] = 666;
}

}
else
{
	if(input[i]==(665+offset2))
{
	vR[i] = 665;
}
else
{
	vR[i] = 664;
}

}

}

}
else
{
	if(input[i]>(659+offset2))
{
	if(input[i]>(661+offset2))
{
	if(input[i]==(663+offset2))
{
	vR[i] = 663;
}
else
{
	vR[i] = 662;
}

}
else
{
	if(input[i]==(661+offset2))
{
	vR[i] = 661;
}
else
{
	vR[i] = 660;
}

}

}
else
{
	if(input[i]>(657+offset2))
{
	if(input[i]==(659+offset2))
{
	vR[i] = 659;
}
else
{
	vR[i] = 658;
}

}
else
{
	if(input[i]==(657+offset2))
{
	vR[i] = 657;
}
else
{
	vR[i] = 656;
}

}

}

}

}
else
{
	if(input[i]>(647+offset2))
{
	if(input[i]>(651+offset2))
{
	if(input[i]>(653+offset2))
{
	if(input[i]==(655+offset2))
{
	vR[i] = 655;
}
else
{
	vR[i] = 654;
}

}
else
{
	if(input[i]==(653+offset2))
{
	vR[i] = 653;
}
else
{
	vR[i] = 652;
}

}

}
else
{
	if(input[i]>(649+offset2))
{
	if(input[i]==(651+offset2))
{
	vR[i] = 651;
}
else
{
	vR[i] = 650;
}

}
else
{
	if(input[i]==(649+offset2))
{
	vR[i] = 649;
}
else
{
	vR[i] = 648;
}

}

}

}
else
{
	if(input[i]>(643+offset2))
{
	if(input[i]>(645+offset2))
{
	if(input[i]==(647+offset2))
{
	vR[i] = 647;
}
else
{
	vR[i] = 646;
}

}
else
{
	if(input[i]==(645+offset2))
{
	vR[i] = 645;
}
else
{
	vR[i] = 644;
}

}

}
else
{
	if(input[i]>(641+offset2))
{
	if(input[i]==(643+offset2))
{
	vR[i] = 643;
}
else
{
	vR[i] = 642;
}

}
else
{
	if(input[i]==(641+offset2))
{
	vR[i] = 641;
}
else
{
	vR[i] = 640;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(575+offset2))
{
	if(input[i]>(607+offset2))
{
	if(input[i]>(623+offset2))
{
	if(input[i]>(631+offset2))
{
	if(input[i]>(635+offset2))
{
	if(input[i]>(637+offset2))
{
	if(input[i]==(639+offset2))
{
	vR[i] = 639;
}
else
{
	vR[i] = 638;
}

}
else
{
	if(input[i]==(637+offset2))
{
	vR[i] = 637;
}
else
{
	vR[i] = 636;
}

}

}
else
{
	if(input[i]>(633+offset2))
{
	if(input[i]==(635+offset2))
{
	vR[i] = 635;
}
else
{
	vR[i] = 634;
}

}
else
{
	if(input[i]==(633+offset2))
{
	vR[i] = 633;
}
else
{
	vR[i] = 632;
}

}

}

}
else
{
	if(input[i]>(627+offset2))
{
	if(input[i]>(629+offset2))
{
	if(input[i]==(631+offset2))
{
	vR[i] = 631;
}
else
{
	vR[i] = 630;
}

}
else
{
	if(input[i]==(629+offset2))
{
	vR[i] = 629;
}
else
{
	vR[i] = 628;
}

}

}
else
{
	if(input[i]>(625+offset2))
{
	if(input[i]==(627+offset2))
{
	vR[i] = 627;
}
else
{
	vR[i] = 626;
}

}
else
{
	if(input[i]==(625+offset2))
{
	vR[i] = 625;
}
else
{
	vR[i] = 624;
}

}

}

}

}
else
{
	if(input[i]>(615+offset2))
{
	if(input[i]>(619+offset2))
{
	if(input[i]>(621+offset2))
{
	if(input[i]==(623+offset2))
{
	vR[i] = 623;
}
else
{
	vR[i] = 622;
}

}
else
{
	if(input[i]==(621+offset2))
{
	vR[i] = 621;
}
else
{
	vR[i] = 620;
}

}

}
else
{
	if(input[i]>(617+offset2))
{
	if(input[i]==(619+offset2))
{
	vR[i] = 619;
}
else
{
	vR[i] = 618;
}

}
else
{
	if(input[i]==(617+offset2))
{
	vR[i] = 617;
}
else
{
	vR[i] = 616;
}

}

}

}
else
{
	if(input[i]>(611+offset2))
{
	if(input[i]>(613+offset2))
{
	if(input[i]==(615+offset2))
{
	vR[i] = 615;
}
else
{
	vR[i] = 614;
}

}
else
{
	if(input[i]==(613+offset2))
{
	vR[i] = 613;
}
else
{
	vR[i] = 612;
}

}

}
else
{
	if(input[i]>(609+offset2))
{
	if(input[i]==(611+offset2))
{
	vR[i] = 611;
}
else
{
	vR[i] = 610;
}

}
else
{
	if(input[i]==(609+offset2))
{
	vR[i] = 609;
}
else
{
	vR[i] = 608;
}

}

}

}

}

}
else
{
	if(input[i]>(591+offset2))
{
	if(input[i]>(599+offset2))
{
	if(input[i]>(603+offset2))
{
	if(input[i]>(605+offset2))
{
	if(input[i]==(607+offset2))
{
	vR[i] = 607;
}
else
{
	vR[i] = 606;
}

}
else
{
	if(input[i]==(605+offset2))
{
	vR[i] = 605;
}
else
{
	vR[i] = 604;
}

}

}
else
{
	if(input[i]>(601+offset2))
{
	if(input[i]==(603+offset2))
{
	vR[i] = 603;
}
else
{
	vR[i] = 602;
}

}
else
{
	if(input[i]==(601+offset2))
{
	vR[i] = 601;
}
else
{
	vR[i] = 600;
}

}

}

}
else
{
	if(input[i]>(595+offset2))
{
	if(input[i]>(597+offset2))
{
	if(input[i]==(599+offset2))
{
	vR[i] = 599;
}
else
{
	vR[i] = 598;
}

}
else
{
	if(input[i]==(597+offset2))
{
	vR[i] = 597;
}
else
{
	vR[i] = 596;
}

}

}
else
{
	if(input[i]>(593+offset2))
{
	if(input[i]==(595+offset2))
{
	vR[i] = 595;
}
else
{
	vR[i] = 594;
}

}
else
{
	if(input[i]==(593+offset2))
{
	vR[i] = 593;
}
else
{
	vR[i] = 592;
}

}

}

}

}
else
{
	if(input[i]>(583+offset2))
{
	if(input[i]>(587+offset2))
{
	if(input[i]>(589+offset2))
{
	if(input[i]==(591+offset2))
{
	vR[i] = 591;
}
else
{
	vR[i] = 590;
}

}
else
{
	if(input[i]==(589+offset2))
{
	vR[i] = 589;
}
else
{
	vR[i] = 588;
}

}

}
else
{
	if(input[i]>(585+offset2))
{
	if(input[i]==(587+offset2))
{
	vR[i] = 587;
}
else
{
	vR[i] = 586;
}

}
else
{
	if(input[i]==(585+offset2))
{
	vR[i] = 585;
}
else
{
	vR[i] = 584;
}

}

}

}
else
{
	if(input[i]>(579+offset2))
{
	if(input[i]>(581+offset2))
{
	if(input[i]==(583+offset2))
{
	vR[i] = 583;
}
else
{
	vR[i] = 582;
}

}
else
{
	if(input[i]==(581+offset2))
{
	vR[i] = 581;
}
else
{
	vR[i] = 580;
}

}

}
else
{
	if(input[i]>(577+offset2))
{
	if(input[i]==(579+offset2))
{
	vR[i] = 579;
}
else
{
	vR[i] = 578;
}

}
else
{
	if(input[i]==(577+offset2))
{
	vR[i] = 577;
}
else
{
	vR[i] = 576;
}

}

}

}

}

}

}
else
{
	if(input[i]>(543+offset2))
{
	if(input[i]>(559+offset2))
{
	if(input[i]>(567+offset2))
{
	if(input[i]>(571+offset2))
{
	if(input[i]>(573+offset2))
{
	if(input[i]==(575+offset2))
{
	vR[i] = 575;
}
else
{
	vR[i] = 574;
}

}
else
{
	if(input[i]==(573+offset2))
{
	vR[i] = 573;
}
else
{
	vR[i] = 572;
}

}

}
else
{
	if(input[i]>(569+offset2))
{
	if(input[i]==(571+offset2))
{
	vR[i] = 571;
}
else
{
	vR[i] = 570;
}

}
else
{
	if(input[i]==(569+offset2))
{
	vR[i] = 569;
}
else
{
	vR[i] = 568;
}

}

}

}
else
{
	if(input[i]>(563+offset2))
{
	if(input[i]>(565+offset2))
{
	if(input[i]==(567+offset2))
{
	vR[i] = 567;
}
else
{
	vR[i] = 566;
}

}
else
{
	if(input[i]==(565+offset2))
{
	vR[i] = 565;
}
else
{
	vR[i] = 564;
}

}

}
else
{
	if(input[i]>(561+offset2))
{
	if(input[i]==(563+offset2))
{
	vR[i] = 563;
}
else
{
	vR[i] = 562;
}

}
else
{
	if(input[i]==(561+offset2))
{
	vR[i] = 561;
}
else
{
	vR[i] = 560;
}

}

}

}

}
else
{
	if(input[i]>(551+offset2))
{
	if(input[i]>(555+offset2))
{
	if(input[i]>(557+offset2))
{
	if(input[i]==(559+offset2))
{
	vR[i] = 559;
}
else
{
	vR[i] = 558;
}

}
else
{
	if(input[i]==(557+offset2))
{
	vR[i] = 557;
}
else
{
	vR[i] = 556;
}

}

}
else
{
	if(input[i]>(553+offset2))
{
	if(input[i]==(555+offset2))
{
	vR[i] = 555;
}
else
{
	vR[i] = 554;
}

}
else
{
	if(input[i]==(553+offset2))
{
	vR[i] = 553;
}
else
{
	vR[i] = 552;
}

}

}

}
else
{
	if(input[i]>(547+offset2))
{
	if(input[i]>(549+offset2))
{
	if(input[i]==(551+offset2))
{
	vR[i] = 551;
}
else
{
	vR[i] = 550;
}

}
else
{
	if(input[i]==(549+offset2))
{
	vR[i] = 549;
}
else
{
	vR[i] = 548;
}

}

}
else
{
	if(input[i]>(545+offset2))
{
	if(input[i]==(547+offset2))
{
	vR[i] = 547;
}
else
{
	vR[i] = 546;
}

}
else
{
	if(input[i]==(545+offset2))
{
	vR[i] = 545;
}
else
{
	vR[i] = 544;
}

}

}

}

}

}
else
{
	if(input[i]>(527+offset2))
{
	if(input[i]>(535+offset2))
{
	if(input[i]>(539+offset2))
{
	if(input[i]>(541+offset2))
{
	if(input[i]==(543+offset2))
{
	vR[i] = 543;
}
else
{
	vR[i] = 542;
}

}
else
{
	if(input[i]==(541+offset2))
{
	vR[i] = 541;
}
else
{
	vR[i] = 540;
}

}

}
else
{
	if(input[i]>(537+offset2))
{
	if(input[i]==(539+offset2))
{
	vR[i] = 539;
}
else
{
	vR[i] = 538;
}

}
else
{
	if(input[i]==(537+offset2))
{
	vR[i] = 537;
}
else
{
	vR[i] = 536;
}

}

}

}
else
{
	if(input[i]>(531+offset2))
{
	if(input[i]>(533+offset2))
{
	if(input[i]==(535+offset2))
{
	vR[i] = 535;
}
else
{
	vR[i] = 534;
}

}
else
{
	if(input[i]==(533+offset2))
{
	vR[i] = 533;
}
else
{
	vR[i] = 532;
}

}

}
else
{
	if(input[i]>(529+offset2))
{
	if(input[i]==(531+offset2))
{
	vR[i] = 531;
}
else
{
	vR[i] = 530;
}

}
else
{
	if(input[i]==(529+offset2))
{
	vR[i] = 529;
}
else
{
	vR[i] = 528;
}

}

}

}

}
else
{
	if(input[i]>(519+offset2))
{
	if(input[i]>(523+offset2))
{
	if(input[i]>(525+offset2))
{
	if(input[i]==(527+offset2))
{
	vR[i] = 527;
}
else
{
	vR[i] = 526;
}

}
else
{
	if(input[i]==(525+offset2))
{
	vR[i] = 525;
}
else
{
	vR[i] = 524;
}

}

}
else
{
	if(input[i]>(521+offset2))
{
	if(input[i]==(523+offset2))
{
	vR[i] = 523;
}
else
{
	vR[i] = 522;
}

}
else
{
	if(input[i]==(521+offset2))
{
	vR[i] = 521;
}
else
{
	vR[i] = 520;
}

}

}

}
else
{
	if(input[i]>(515+offset2))
{
	if(input[i]>(517+offset2))
{
	if(input[i]==(519+offset2))
{
	vR[i] = 519;
}
else
{
	vR[i] = 518;
}

}
else
{
	if(input[i]==(517+offset2))
{
	vR[i] = 517;
}
else
{
	vR[i] = 516;
}

}

}
else
{
	if(input[i]>(513+offset2))
{
	if(input[i]==(515+offset2))
{
	vR[i] = 515;
}
else
{
	vR[i] = 514;
}

}
else
{
	if(input[i]==(513+offset2))
{
	vR[i] = 513;
}
else
{
	vR[i] = 512;
}

}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(255+offset2))
{
	if(input[i]>(383+offset2))
{
	if(input[i]>(447+offset2))
{
	if(input[i]>(479+offset2))
{
	if(input[i]>(495+offset2))
{
	if(input[i]>(503+offset2))
{
	if(input[i]>(507+offset2))
{
	if(input[i]>(509+offset2))
{
	if(input[i]==(511+offset2))
{
	vR[i] = 511;
}
else
{
	vR[i] = 510;
}

}
else
{
	if(input[i]==(509+offset2))
{
	vR[i] = 509;
}
else
{
	vR[i] = 508;
}

}

}
else
{
	if(input[i]>(505+offset2))
{
	if(input[i]==(507+offset2))
{
	vR[i] = 507;
}
else
{
	vR[i] = 506;
}

}
else
{
	if(input[i]==(505+offset2))
{
	vR[i] = 505;
}
else
{
	vR[i] = 504;
}

}

}

}
else
{
	if(input[i]>(499+offset2))
{
	if(input[i]>(501+offset2))
{
	if(input[i]==(503+offset2))
{
	vR[i] = 503;
}
else
{
	vR[i] = 502;
}

}
else
{
	if(input[i]==(501+offset2))
{
	vR[i] = 501;
}
else
{
	vR[i] = 500;
}

}

}
else
{
	if(input[i]>(497+offset2))
{
	if(input[i]==(499+offset2))
{
	vR[i] = 499;
}
else
{
	vR[i] = 498;
}

}
else
{
	if(input[i]==(497+offset2))
{
	vR[i] = 497;
}
else
{
	vR[i] = 496;
}

}

}

}

}
else
{
	if(input[i]>(487+offset2))
{
	if(input[i]>(491+offset2))
{
	if(input[i]>(493+offset2))
{
	if(input[i]==(495+offset2))
{
	vR[i] = 495;
}
else
{
	vR[i] = 494;
}

}
else
{
	if(input[i]==(493+offset2))
{
	vR[i] = 493;
}
else
{
	vR[i] = 492;
}

}

}
else
{
	if(input[i]>(489+offset2))
{
	if(input[i]==(491+offset2))
{
	vR[i] = 491;
}
else
{
	vR[i] = 490;
}

}
else
{
	if(input[i]==(489+offset2))
{
	vR[i] = 489;
}
else
{
	vR[i] = 488;
}

}

}

}
else
{
	if(input[i]>(483+offset2))
{
	if(input[i]>(485+offset2))
{
	if(input[i]==(487+offset2))
{
	vR[i] = 487;
}
else
{
	vR[i] = 486;
}

}
else
{
	if(input[i]==(485+offset2))
{
	vR[i] = 485;
}
else
{
	vR[i] = 484;
}

}

}
else
{
	if(input[i]>(481+offset2))
{
	if(input[i]==(483+offset2))
{
	vR[i] = 483;
}
else
{
	vR[i] = 482;
}

}
else
{
	if(input[i]==(481+offset2))
{
	vR[i] = 481;
}
else
{
	vR[i] = 480;
}

}

}

}

}

}
else
{
	if(input[i]>(463+offset2))
{
	if(input[i]>(471+offset2))
{
	if(input[i]>(475+offset2))
{
	if(input[i]>(477+offset2))
{
	if(input[i]==(479+offset2))
{
	vR[i] = 479;
}
else
{
	vR[i] = 478;
}

}
else
{
	if(input[i]==(477+offset2))
{
	vR[i] = 477;
}
else
{
	vR[i] = 476;
}

}

}
else
{
	if(input[i]>(473+offset2))
{
	if(input[i]==(475+offset2))
{
	vR[i] = 475;
}
else
{
	vR[i] = 474;
}

}
else
{
	if(input[i]==(473+offset2))
{
	vR[i] = 473;
}
else
{
	vR[i] = 472;
}

}

}

}
else
{
	if(input[i]>(467+offset2))
{
	if(input[i]>(469+offset2))
{
	if(input[i]==(471+offset2))
{
	vR[i] = 471;
}
else
{
	vR[i] = 470;
}

}
else
{
	if(input[i]==(469+offset2))
{
	vR[i] = 469;
}
else
{
	vR[i] = 468;
}

}

}
else
{
	if(input[i]>(465+offset2))
{
	if(input[i]==(467+offset2))
{
	vR[i] = 467;
}
else
{
	vR[i] = 466;
}

}
else
{
	if(input[i]==(465+offset2))
{
	vR[i] = 465;
}
else
{
	vR[i] = 464;
}

}

}

}

}
else
{
	if(input[i]>(455+offset2))
{
	if(input[i]>(459+offset2))
{
	if(input[i]>(461+offset2))
{
	if(input[i]==(463+offset2))
{
	vR[i] = 463;
}
else
{
	vR[i] = 462;
}

}
else
{
	if(input[i]==(461+offset2))
{
	vR[i] = 461;
}
else
{
	vR[i] = 460;
}

}

}
else
{
	if(input[i]>(457+offset2))
{
	if(input[i]==(459+offset2))
{
	vR[i] = 459;
}
else
{
	vR[i] = 458;
}

}
else
{
	if(input[i]==(457+offset2))
{
	vR[i] = 457;
}
else
{
	vR[i] = 456;
}

}

}

}
else
{
	if(input[i]>(451+offset2))
{
	if(input[i]>(453+offset2))
{
	if(input[i]==(455+offset2))
{
	vR[i] = 455;
}
else
{
	vR[i] = 454;
}

}
else
{
	if(input[i]==(453+offset2))
{
	vR[i] = 453;
}
else
{
	vR[i] = 452;
}

}

}
else
{
	if(input[i]>(449+offset2))
{
	if(input[i]==(451+offset2))
{
	vR[i] = 451;
}
else
{
	vR[i] = 450;
}

}
else
{
	if(input[i]==(449+offset2))
{
	vR[i] = 449;
}
else
{
	vR[i] = 448;
}

}

}

}

}

}

}
else
{
	if(input[i]>(415+offset2))
{
	if(input[i]>(431+offset2))
{
	if(input[i]>(439+offset2))
{
	if(input[i]>(443+offset2))
{
	if(input[i]>(445+offset2))
{
	if(input[i]==(447+offset2))
{
	vR[i] = 447;
}
else
{
	vR[i] = 446;
}

}
else
{
	if(input[i]==(445+offset2))
{
	vR[i] = 445;
}
else
{
	vR[i] = 444;
}

}

}
else
{
	if(input[i]>(441+offset2))
{
	if(input[i]==(443+offset2))
{
	vR[i] = 443;
}
else
{
	vR[i] = 442;
}

}
else
{
	if(input[i]==(441+offset2))
{
	vR[i] = 441;
}
else
{
	vR[i] = 440;
}

}

}

}
else
{
	if(input[i]>(435+offset2))
{
	if(input[i]>(437+offset2))
{
	if(input[i]==(439+offset2))
{
	vR[i] = 439;
}
else
{
	vR[i] = 438;
}

}
else
{
	if(input[i]==(437+offset2))
{
	vR[i] = 437;
}
else
{
	vR[i] = 436;
}

}

}
else
{
	if(input[i]>(433+offset2))
{
	if(input[i]==(435+offset2))
{
	vR[i] = 435;
}
else
{
	vR[i] = 434;
}

}
else
{
	if(input[i]==(433+offset2))
{
	vR[i] = 433;
}
else
{
	vR[i] = 432;
}

}

}

}

}
else
{
	if(input[i]>(423+offset2))
{
	if(input[i]>(427+offset2))
{
	if(input[i]>(429+offset2))
{
	if(input[i]==(431+offset2))
{
	vR[i] = 431;
}
else
{
	vR[i] = 430;
}

}
else
{
	if(input[i]==(429+offset2))
{
	vR[i] = 429;
}
else
{
	vR[i] = 428;
}

}

}
else
{
	if(input[i]>(425+offset2))
{
	if(input[i]==(427+offset2))
{
	vR[i] = 427;
}
else
{
	vR[i] = 426;
}

}
else
{
	if(input[i]==(425+offset2))
{
	vR[i] = 425;
}
else
{
	vR[i] = 424;
}

}

}

}
else
{
	if(input[i]>(419+offset2))
{
	if(input[i]>(421+offset2))
{
	if(input[i]==(423+offset2))
{
	vR[i] = 423;
}
else
{
	vR[i] = 422;
}

}
else
{
	if(input[i]==(421+offset2))
{
	vR[i] = 421;
}
else
{
	vR[i] = 420;
}

}

}
else
{
	if(input[i]>(417+offset2))
{
	if(input[i]==(419+offset2))
{
	vR[i] = 419;
}
else
{
	vR[i] = 418;
}

}
else
{
	if(input[i]==(417+offset2))
{
	vR[i] = 417;
}
else
{
	vR[i] = 416;
}

}

}

}

}

}
else
{
	if(input[i]>(399+offset2))
{
	if(input[i]>(407+offset2))
{
	if(input[i]>(411+offset2))
{
	if(input[i]>(413+offset2))
{
	if(input[i]==(415+offset2))
{
	vR[i] = 415;
}
else
{
	vR[i] = 414;
}

}
else
{
	if(input[i]==(413+offset2))
{
	vR[i] = 413;
}
else
{
	vR[i] = 412;
}

}

}
else
{
	if(input[i]>(409+offset2))
{
	if(input[i]==(411+offset2))
{
	vR[i] = 411;
}
else
{
	vR[i] = 410;
}

}
else
{
	if(input[i]==(409+offset2))
{
	vR[i] = 409;
}
else
{
	vR[i] = 408;
}

}

}

}
else
{
	if(input[i]>(403+offset2))
{
	if(input[i]>(405+offset2))
{
	if(input[i]==(407+offset2))
{
	vR[i] = 407;
}
else
{
	vR[i] = 406;
}

}
else
{
	if(input[i]==(405+offset2))
{
	vR[i] = 405;
}
else
{
	vR[i] = 404;
}

}

}
else
{
	if(input[i]>(401+offset2))
{
	if(input[i]==(403+offset2))
{
	vR[i] = 403;
}
else
{
	vR[i] = 402;
}

}
else
{
	if(input[i]==(401+offset2))
{
	vR[i] = 401;
}
else
{
	vR[i] = 400;
}

}

}

}

}
else
{
	if(input[i]>(391+offset2))
{
	if(input[i]>(395+offset2))
{
	if(input[i]>(397+offset2))
{
	if(input[i]==(399+offset2))
{
	vR[i] = 399;
}
else
{
	vR[i] = 398;
}

}
else
{
	if(input[i]==(397+offset2))
{
	vR[i] = 397;
}
else
{
	vR[i] = 396;
}

}

}
else
{
	if(input[i]>(393+offset2))
{
	if(input[i]==(395+offset2))
{
	vR[i] = 395;
}
else
{
	vR[i] = 394;
}

}
else
{
	if(input[i]==(393+offset2))
{
	vR[i] = 393;
}
else
{
	vR[i] = 392;
}

}

}

}
else
{
	if(input[i]>(387+offset2))
{
	if(input[i]>(389+offset2))
{
	if(input[i]==(391+offset2))
{
	vR[i] = 391;
}
else
{
	vR[i] = 390;
}

}
else
{
	if(input[i]==(389+offset2))
{
	vR[i] = 389;
}
else
{
	vR[i] = 388;
}

}

}
else
{
	if(input[i]>(385+offset2))
{
	if(input[i]==(387+offset2))
{
	vR[i] = 387;
}
else
{
	vR[i] = 386;
}

}
else
{
	if(input[i]==(385+offset2))
{
	vR[i] = 385;
}
else
{
	vR[i] = 384;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(319+offset2))
{
	if(input[i]>(351+offset2))
{
	if(input[i]>(367+offset2))
{
	if(input[i]>(375+offset2))
{
	if(input[i]>(379+offset2))
{
	if(input[i]>(381+offset2))
{
	if(input[i]==(383+offset2))
{
	vR[i] = 383;
}
else
{
	vR[i] = 382;
}

}
else
{
	if(input[i]==(381+offset2))
{
	vR[i] = 381;
}
else
{
	vR[i] = 380;
}

}

}
else
{
	if(input[i]>(377+offset2))
{
	if(input[i]==(379+offset2))
{
	vR[i] = 379;
}
else
{
	vR[i] = 378;
}

}
else
{
	if(input[i]==(377+offset2))
{
	vR[i] = 377;
}
else
{
	vR[i] = 376;
}

}

}

}
else
{
	if(input[i]>(371+offset2))
{
	if(input[i]>(373+offset2))
{
	if(input[i]==(375+offset2))
{
	vR[i] = 375;
}
else
{
	vR[i] = 374;
}

}
else
{
	if(input[i]==(373+offset2))
{
	vR[i] = 373;
}
else
{
	vR[i] = 372;
}

}

}
else
{
	if(input[i]>(369+offset2))
{
	if(input[i]==(371+offset2))
{
	vR[i] = 371;
}
else
{
	vR[i] = 370;
}

}
else
{
	if(input[i]==(369+offset2))
{
	vR[i] = 369;
}
else
{
	vR[i] = 368;
}

}

}

}

}
else
{
	if(input[i]>(359+offset2))
{
	if(input[i]>(363+offset2))
{
	if(input[i]>(365+offset2))
{
	if(input[i]==(367+offset2))
{
	vR[i] = 367;
}
else
{
	vR[i] = 366;
}

}
else
{
	if(input[i]==(365+offset2))
{
	vR[i] = 365;
}
else
{
	vR[i] = 364;
}

}

}
else
{
	if(input[i]>(361+offset2))
{
	if(input[i]==(363+offset2))
{
	vR[i] = 363;
}
else
{
	vR[i] = 362;
}

}
else
{
	if(input[i]==(361+offset2))
{
	vR[i] = 361;
}
else
{
	vR[i] = 360;
}

}

}

}
else
{
	if(input[i]>(355+offset2))
{
	if(input[i]>(357+offset2))
{
	if(input[i]==(359+offset2))
{
	vR[i] = 359;
}
else
{
	vR[i] = 358;
}

}
else
{
	if(input[i]==(357+offset2))
{
	vR[i] = 357;
}
else
{
	vR[i] = 356;
}

}

}
else
{
	if(input[i]>(353+offset2))
{
	if(input[i]==(355+offset2))
{
	vR[i] = 355;
}
else
{
	vR[i] = 354;
}

}
else
{
	if(input[i]==(353+offset2))
{
	vR[i] = 353;
}
else
{
	vR[i] = 352;
}

}

}

}

}

}
else
{
	if(input[i]>(335+offset2))
{
	if(input[i]>(343+offset2))
{
	if(input[i]>(347+offset2))
{
	if(input[i]>(349+offset2))
{
	if(input[i]==(351+offset2))
{
	vR[i] = 351;
}
else
{
	vR[i] = 350;
}

}
else
{
	if(input[i]==(349+offset2))
{
	vR[i] = 349;
}
else
{
	vR[i] = 348;
}

}

}
else
{
	if(input[i]>(345+offset2))
{
	if(input[i]==(347+offset2))
{
	vR[i] = 347;
}
else
{
	vR[i] = 346;
}

}
else
{
	if(input[i]==(345+offset2))
{
	vR[i] = 345;
}
else
{
	vR[i] = 344;
}

}

}

}
else
{
	if(input[i]>(339+offset2))
{
	if(input[i]>(341+offset2))
{
	if(input[i]==(343+offset2))
{
	vR[i] = 343;
}
else
{
	vR[i] = 342;
}

}
else
{
	if(input[i]==(341+offset2))
{
	vR[i] = 341;
}
else
{
	vR[i] = 340;
}

}

}
else
{
	if(input[i]>(337+offset2))
{
	if(input[i]==(339+offset2))
{
	vR[i] = 339;
}
else
{
	vR[i] = 338;
}

}
else
{
	if(input[i]==(337+offset2))
{
	vR[i] = 337;
}
else
{
	vR[i] = 336;
}

}

}

}

}
else
{
	if(input[i]>(327+offset2))
{
	if(input[i]>(331+offset2))
{
	if(input[i]>(333+offset2))
{
	if(input[i]==(335+offset2))
{
	vR[i] = 335;
}
else
{
	vR[i] = 334;
}

}
else
{
	if(input[i]==(333+offset2))
{
	vR[i] = 333;
}
else
{
	vR[i] = 332;
}

}

}
else
{
	if(input[i]>(329+offset2))
{
	if(input[i]==(331+offset2))
{
	vR[i] = 331;
}
else
{
	vR[i] = 330;
}

}
else
{
	if(input[i]==(329+offset2))
{
	vR[i] = 329;
}
else
{
	vR[i] = 328;
}

}

}

}
else
{
	if(input[i]>(323+offset2))
{
	if(input[i]>(325+offset2))
{
	if(input[i]==(327+offset2))
{
	vR[i] = 327;
}
else
{
	vR[i] = 326;
}

}
else
{
	if(input[i]==(325+offset2))
{
	vR[i] = 325;
}
else
{
	vR[i] = 324;
}

}

}
else
{
	if(input[i]>(321+offset2))
{
	if(input[i]==(323+offset2))
{
	vR[i] = 323;
}
else
{
	vR[i] = 322;
}

}
else
{
	if(input[i]==(321+offset2))
{
	vR[i] = 321;
}
else
{
	vR[i] = 320;
}

}

}

}

}

}

}
else
{
	if(input[i]>(287+offset2))
{
	if(input[i]>(303+offset2))
{
	if(input[i]>(311+offset2))
{
	if(input[i]>(315+offset2))
{
	if(input[i]>(317+offset2))
{
	if(input[i]==(319+offset2))
{
	vR[i] = 319;
}
else
{
	vR[i] = 318;
}

}
else
{
	if(input[i]==(317+offset2))
{
	vR[i] = 317;
}
else
{
	vR[i] = 316;
}

}

}
else
{
	if(input[i]>(313+offset2))
{
	if(input[i]==(315+offset2))
{
	vR[i] = 315;
}
else
{
	vR[i] = 314;
}

}
else
{
	if(input[i]==(313+offset2))
{
	vR[i] = 313;
}
else
{
	vR[i] = 312;
}

}

}

}
else
{
	if(input[i]>(307+offset2))
{
	if(input[i]>(309+offset2))
{
	if(input[i]==(311+offset2))
{
	vR[i] = 311;
}
else
{
	vR[i] = 310;
}

}
else
{
	if(input[i]==(309+offset2))
{
	vR[i] = 309;
}
else
{
	vR[i] = 308;
}

}

}
else
{
	if(input[i]>(305+offset2))
{
	if(input[i]==(307+offset2))
{
	vR[i] = 307;
}
else
{
	vR[i] = 306;
}

}
else
{
	if(input[i]==(305+offset2))
{
	vR[i] = 305;
}
else
{
	vR[i] = 304;
}

}

}

}

}
else
{
	if(input[i]>(295+offset2))
{
	if(input[i]>(299+offset2))
{
	if(input[i]>(301+offset2))
{
	if(input[i]==(303+offset2))
{
	vR[i] = 303;
}
else
{
	vR[i] = 302;
}

}
else
{
	if(input[i]==(301+offset2))
{
	vR[i] = 301;
}
else
{
	vR[i] = 300;
}

}

}
else
{
	if(input[i]>(297+offset2))
{
	if(input[i]==(299+offset2))
{
	vR[i] = 299;
}
else
{
	vR[i] = 298;
}

}
else
{
	if(input[i]==(297+offset2))
{
	vR[i] = 297;
}
else
{
	vR[i] = 296;
}

}

}

}
else
{
	if(input[i]>(291+offset2))
{
	if(input[i]>(293+offset2))
{
	if(input[i]==(295+offset2))
{
	vR[i] = 295;
}
else
{
	vR[i] = 294;
}

}
else
{
	if(input[i]==(293+offset2))
{
	vR[i] = 293;
}
else
{
	vR[i] = 292;
}

}

}
else
{
	if(input[i]>(289+offset2))
{
	if(input[i]==(291+offset2))
{
	vR[i] = 291;
}
else
{
	vR[i] = 290;
}

}
else
{
	if(input[i]==(289+offset2))
{
	vR[i] = 289;
}
else
{
	vR[i] = 288;
}

}

}

}

}

}
else
{
	if(input[i]>(271+offset2))
{
	if(input[i]>(279+offset2))
{
	if(input[i]>(283+offset2))
{
	if(input[i]>(285+offset2))
{
	if(input[i]==(287+offset2))
{
	vR[i] = 287;
}
else
{
	vR[i] = 286;
}

}
else
{
	if(input[i]==(285+offset2))
{
	vR[i] = 285;
}
else
{
	vR[i] = 284;
}

}

}
else
{
	if(input[i]>(281+offset2))
{
	if(input[i]==(283+offset2))
{
	vR[i] = 283;
}
else
{
	vR[i] = 282;
}

}
else
{
	if(input[i]==(281+offset2))
{
	vR[i] = 281;
}
else
{
	vR[i] = 280;
}

}

}

}
else
{
	if(input[i]>(275+offset2))
{
	if(input[i]>(277+offset2))
{
	if(input[i]==(279+offset2))
{
	vR[i] = 279;
}
else
{
	vR[i] = 278;
}

}
else
{
	if(input[i]==(277+offset2))
{
	vR[i] = 277;
}
else
{
	vR[i] = 276;
}

}

}
else
{
	if(input[i]>(273+offset2))
{
	if(input[i]==(275+offset2))
{
	vR[i] = 275;
}
else
{
	vR[i] = 274;
}

}
else
{
	if(input[i]==(273+offset2))
{
	vR[i] = 273;
}
else
{
	vR[i] = 272;
}

}

}

}

}
else
{
	if(input[i]>(263+offset2))
{
	if(input[i]>(267+offset2))
{
	if(input[i]>(269+offset2))
{
	if(input[i]==(271+offset2))
{
	vR[i] = 271;
}
else
{
	vR[i] = 270;
}

}
else
{
	if(input[i]==(269+offset2))
{
	vR[i] = 269;
}
else
{
	vR[i] = 268;
}

}

}
else
{
	if(input[i]>(265+offset2))
{
	if(input[i]==(267+offset2))
{
	vR[i] = 267;
}
else
{
	vR[i] = 266;
}

}
else
{
	if(input[i]==(265+offset2))
{
	vR[i] = 265;
}
else
{
	vR[i] = 264;
}

}

}

}
else
{
	if(input[i]>(259+offset2))
{
	if(input[i]>(261+offset2))
{
	if(input[i]==(263+offset2))
{
	vR[i] = 263;
}
else
{
	vR[i] = 262;
}

}
else
{
	if(input[i]==(261+offset2))
{
	vR[i] = 261;
}
else
{
	vR[i] = 260;
}

}

}
else
{
	if(input[i]>(257+offset2))
{
	if(input[i]==(259+offset2))
{
	vR[i] = 259;
}
else
{
	vR[i] = 258;
}

}
else
{
	if(input[i]==(257+offset2))
{
	vR[i] = 257;
}
else
{
	vR[i] = 256;
}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(127+offset2))
{
	if(input[i]>(191+offset2))
{
	if(input[i]>(223+offset2))
{
	if(input[i]>(239+offset2))
{
	if(input[i]>(247+offset2))
{
	if(input[i]>(251+offset2))
{
	if(input[i]>(253+offset2))
{
	if(input[i]==(255+offset2))
{
	vR[i] = 255;
}
else
{
	vR[i] = 254;
}

}
else
{
	if(input[i]==(253+offset2))
{
	vR[i] = 253;
}
else
{
	vR[i] = 252;
}

}

}
else
{
	if(input[i]>(249+offset2))
{
	if(input[i]==(251+offset2))
{
	vR[i] = 251;
}
else
{
	vR[i] = 250;
}

}
else
{
	if(input[i]==(249+offset2))
{
	vR[i] = 249;
}
else
{
	vR[i] = 248;
}

}

}

}
else
{
	if(input[i]>(243+offset2))
{
	if(input[i]>(245+offset2))
{
	if(input[i]==(247+offset2))
{
	vR[i] = 247;
}
else
{
	vR[i] = 246;
}

}
else
{
	if(input[i]==(245+offset2))
{
	vR[i] = 245;
}
else
{
	vR[i] = 244;
}

}

}
else
{
	if(input[i]>(241+offset2))
{
	if(input[i]==(243+offset2))
{
	vR[i] = 243;
}
else
{
	vR[i] = 242;
}

}
else
{
	if(input[i]==(241+offset2))
{
	vR[i] = 241;
}
else
{
	vR[i] = 240;
}

}

}

}

}
else
{
	if(input[i]>(231+offset2))
{
	if(input[i]>(235+offset2))
{
	if(input[i]>(237+offset2))
{
	if(input[i]==(239+offset2))
{
	vR[i] = 239;
}
else
{
	vR[i] = 238;
}

}
else
{
	if(input[i]==(237+offset2))
{
	vR[i] = 237;
}
else
{
	vR[i] = 236;
}

}

}
else
{
	if(input[i]>(233+offset2))
{
	if(input[i]==(235+offset2))
{
	vR[i] = 235;
}
else
{
	vR[i] = 234;
}

}
else
{
	if(input[i]==(233+offset2))
{
	vR[i] = 233;
}
else
{
	vR[i] = 232;
}

}

}

}
else
{
	if(input[i]>(227+offset2))
{
	if(input[i]>(229+offset2))
{
	if(input[i]==(231+offset2))
{
	vR[i] = 231;
}
else
{
	vR[i] = 230;
}

}
else
{
	if(input[i]==(229+offset2))
{
	vR[i] = 229;
}
else
{
	vR[i] = 228;
}

}

}
else
{
	if(input[i]>(225+offset2))
{
	if(input[i]==(227+offset2))
{
	vR[i] = 227;
}
else
{
	vR[i] = 226;
}

}
else
{
	if(input[i]==(225+offset2))
{
	vR[i] = 225;
}
else
{
	vR[i] = 224;
}

}

}

}

}

}
else
{
	if(input[i]>(207+offset2))
{
	if(input[i]>(215+offset2))
{
	if(input[i]>(219+offset2))
{
	if(input[i]>(221+offset2))
{
	if(input[i]==(223+offset2))
{
	vR[i] = 223;
}
else
{
	vR[i] = 222;
}

}
else
{
	if(input[i]==(221+offset2))
{
	vR[i] = 221;
}
else
{
	vR[i] = 220;
}

}

}
else
{
	if(input[i]>(217+offset2))
{
	if(input[i]==(219+offset2))
{
	vR[i] = 219;
}
else
{
	vR[i] = 218;
}

}
else
{
	if(input[i]==(217+offset2))
{
	vR[i] = 217;
}
else
{
	vR[i] = 216;
}

}

}

}
else
{
	if(input[i]>(211+offset2))
{
	if(input[i]>(213+offset2))
{
	if(input[i]==(215+offset2))
{
	vR[i] = 215;
}
else
{
	vR[i] = 214;
}

}
else
{
	if(input[i]==(213+offset2))
{
	vR[i] = 213;
}
else
{
	vR[i] = 212;
}

}

}
else
{
	if(input[i]>(209+offset2))
{
	if(input[i]==(211+offset2))
{
	vR[i] = 211;
}
else
{
	vR[i] = 210;
}

}
else
{
	if(input[i]==(209+offset2))
{
	vR[i] = 209;
}
else
{
	vR[i] = 208;
}

}

}

}

}
else
{
	if(input[i]>(199+offset2))
{
	if(input[i]>(203+offset2))
{
	if(input[i]>(205+offset2))
{
	if(input[i]==(207+offset2))
{
	vR[i] = 207;
}
else
{
	vR[i] = 206;
}

}
else
{
	if(input[i]==(205+offset2))
{
	vR[i] = 205;
}
else
{
	vR[i] = 204;
}

}

}
else
{
	if(input[i]>(201+offset2))
{
	if(input[i]==(203+offset2))
{
	vR[i] = 203;
}
else
{
	vR[i] = 202;
}

}
else
{
	if(input[i]==(201+offset2))
{
	vR[i] = 201;
}
else
{
	vR[i] = 200;
}

}

}

}
else
{
	if(input[i]>(195+offset2))
{
	if(input[i]>(197+offset2))
{
	if(input[i]==(199+offset2))
{
	vR[i] = 199;
}
else
{
	vR[i] = 198;
}

}
else
{
	if(input[i]==(197+offset2))
{
	vR[i] = 197;
}
else
{
	vR[i] = 196;
}

}

}
else
{
	if(input[i]>(193+offset2))
{
	if(input[i]==(195+offset2))
{
	vR[i] = 195;
}
else
{
	vR[i] = 194;
}

}
else
{
	if(input[i]==(193+offset2))
{
	vR[i] = 193;
}
else
{
	vR[i] = 192;
}

}

}

}

}

}

}
else
{
	if(input[i]>(159+offset2))
{
	if(input[i]>(175+offset2))
{
	if(input[i]>(183+offset2))
{
	if(input[i]>(187+offset2))
{
	if(input[i]>(189+offset2))
{
	if(input[i]==(191+offset2))
{
	vR[i] = 191;
}
else
{
	vR[i] = 190;
}

}
else
{
	if(input[i]==(189+offset2))
{
	vR[i] = 189;
}
else
{
	vR[i] = 188;
}

}

}
else
{
	if(input[i]>(185+offset2))
{
	if(input[i]==(187+offset2))
{
	vR[i] = 187;
}
else
{
	vR[i] = 186;
}

}
else
{
	if(input[i]==(185+offset2))
{
	vR[i] = 185;
}
else
{
	vR[i] = 184;
}

}

}

}
else
{
	if(input[i]>(179+offset2))
{
	if(input[i]>(181+offset2))
{
	if(input[i]==(183+offset2))
{
	vR[i] = 183;
}
else
{
	vR[i] = 182;
}

}
else
{
	if(input[i]==(181+offset2))
{
	vR[i] = 181;
}
else
{
	vR[i] = 180;
}

}

}
else
{
	if(input[i]>(177+offset2))
{
	if(input[i]==(179+offset2))
{
	vR[i] = 179;
}
else
{
	vR[i] = 178;
}

}
else
{
	if(input[i]==(177+offset2))
{
	vR[i] = 177;
}
else
{
	vR[i] = 176;
}

}

}

}

}
else
{
	if(input[i]>(167+offset2))
{
	if(input[i]>(171+offset2))
{
	if(input[i]>(173+offset2))
{
	if(input[i]==(175+offset2))
{
	vR[i] = 175;
}
else
{
	vR[i] = 174;
}

}
else
{
	if(input[i]==(173+offset2))
{
	vR[i] = 173;
}
else
{
	vR[i] = 172;
}

}

}
else
{
	if(input[i]>(169+offset2))
{
	if(input[i]==(171+offset2))
{
	vR[i] = 171;
}
else
{
	vR[i] = 170;
}

}
else
{
	if(input[i]==(169+offset2))
{
	vR[i] = 169;
}
else
{
	vR[i] = 168;
}

}

}

}
else
{
	if(input[i]>(163+offset2))
{
	if(input[i]>(165+offset2))
{
	if(input[i]==(167+offset2))
{
	vR[i] = 167;
}
else
{
	vR[i] = 166;
}

}
else
{
	if(input[i]==(165+offset2))
{
	vR[i] = 165;
}
else
{
	vR[i] = 164;
}

}

}
else
{
	if(input[i]>(161+offset2))
{
	if(input[i]==(163+offset2))
{
	vR[i] = 163;
}
else
{
	vR[i] = 162;
}

}
else
{
	if(input[i]==(161+offset2))
{
	vR[i] = 161;
}
else
{
	vR[i] = 160;
}

}

}

}

}

}
else
{
	if(input[i]>(143+offset2))
{
	if(input[i]>(151+offset2))
{
	if(input[i]>(155+offset2))
{
	if(input[i]>(157+offset2))
{
	if(input[i]==(159+offset2))
{
	vR[i] = 159;
}
else
{
	vR[i] = 158;
}

}
else
{
	if(input[i]==(157+offset2))
{
	vR[i] = 157;
}
else
{
	vR[i] = 156;
}

}

}
else
{
	if(input[i]>(153+offset2))
{
	if(input[i]==(155+offset2))
{
	vR[i] = 155;
}
else
{
	vR[i] = 154;
}

}
else
{
	if(input[i]==(153+offset2))
{
	vR[i] = 153;
}
else
{
	vR[i] = 152;
}

}

}

}
else
{
	if(input[i]>(147+offset2))
{
	if(input[i]>(149+offset2))
{
	if(input[i]==(151+offset2))
{
	vR[i] = 151;
}
else
{
	vR[i] = 150;
}

}
else
{
	if(input[i]==(149+offset2))
{
	vR[i] = 149;
}
else
{
	vR[i] = 148;
}

}

}
else
{
	if(input[i]>(145+offset2))
{
	if(input[i]==(147+offset2))
{
	vR[i] = 147;
}
else
{
	vR[i] = 146;
}

}
else
{
	if(input[i]==(145+offset2))
{
	vR[i] = 145;
}
else
{
	vR[i] = 144;
}

}

}

}

}
else
{
	if(input[i]>(135+offset2))
{
	if(input[i]>(139+offset2))
{
	if(input[i]>(141+offset2))
{
	if(input[i]==(143+offset2))
{
	vR[i] = 143;
}
else
{
	vR[i] = 142;
}

}
else
{
	if(input[i]==(141+offset2))
{
	vR[i] = 141;
}
else
{
	vR[i] = 140;
}

}

}
else
{
	if(input[i]>(137+offset2))
{
	if(input[i]==(139+offset2))
{
	vR[i] = 139;
}
else
{
	vR[i] = 138;
}

}
else
{
	if(input[i]==(137+offset2))
{
	vR[i] = 137;
}
else
{
	vR[i] = 136;
}

}

}

}
else
{
	if(input[i]>(131+offset2))
{
	if(input[i]>(133+offset2))
{
	if(input[i]==(135+offset2))
{
	vR[i] = 135;
}
else
{
	vR[i] = 134;
}

}
else
{
	if(input[i]==(133+offset2))
{
	vR[i] = 133;
}
else
{
	vR[i] = 132;
}

}

}
else
{
	if(input[i]>(129+offset2))
{
	if(input[i]==(131+offset2))
{
	vR[i] = 131;
}
else
{
	vR[i] = 130;
}

}
else
{
	if(input[i]==(129+offset2))
{
	vR[i] = 129;
}
else
{
	vR[i] = 128;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(63+offset2))
{
	if(input[i]>(95+offset2))
{
	if(input[i]>(111+offset2))
{
	if(input[i]>(119+offset2))
{
	if(input[i]>(123+offset2))
{
	if(input[i]>(125+offset2))
{
	if(input[i]==(127+offset2))
{
	vR[i] = 127;
}
else
{
	vR[i] = 126;
}

}
else
{
	if(input[i]==(125+offset2))
{
	vR[i] = 125;
}
else
{
	vR[i] = 124;
}

}

}
else
{
	if(input[i]>(121+offset2))
{
	if(input[i]==(123+offset2))
{
	vR[i] = 123;
}
else
{
	vR[i] = 122;
}

}
else
{
	if(input[i]==(121+offset2))
{
	vR[i] = 121;
}
else
{
	vR[i] = 120;
}

}

}

}
else
{
	if(input[i]>(115+offset2))
{
	if(input[i]>(117+offset2))
{
	if(input[i]==(119+offset2))
{
	vR[i] = 119;
}
else
{
	vR[i] = 118;
}

}
else
{
	if(input[i]==(117+offset2))
{
	vR[i] = 117;
}
else
{
	vR[i] = 116;
}

}

}
else
{
	if(input[i]>(113+offset2))
{
	if(input[i]==(115+offset2))
{
	vR[i] = 115;
}
else
{
	vR[i] = 114;
}

}
else
{
	if(input[i]==(113+offset2))
{
	vR[i] = 113;
}
else
{
	vR[i] = 112;
}

}

}

}

}
else
{
	if(input[i]>(103+offset2))
{
	if(input[i]>(107+offset2))
{
	if(input[i]>(109+offset2))
{
	if(input[i]==(111+offset2))
{
	vR[i] = 111;
}
else
{
	vR[i] = 110;
}

}
else
{
	if(input[i]==(109+offset2))
{
	vR[i] = 109;
}
else
{
	vR[i] = 108;
}

}

}
else
{
	if(input[i]>(105+offset2))
{
	if(input[i]==(107+offset2))
{
	vR[i] = 107;
}
else
{
	vR[i] = 106;
}

}
else
{
	if(input[i]==(105+offset2))
{
	vR[i] = 105;
}
else
{
	vR[i] = 104;
}

}

}

}
else
{
	if(input[i]>(99+offset2))
{
	if(input[i]>(101+offset2))
{
	if(input[i]==(103+offset2))
{
	vR[i] = 103;
}
else
{
	vR[i] = 102;
}

}
else
{
	if(input[i]==(101+offset2))
{
	vR[i] = 101;
}
else
{
	vR[i] = 100;
}

}

}
else
{
	if(input[i]>(97+offset2))
{
	if(input[i]==(99+offset2))
{
	vR[i] = 99;
}
else
{
	vR[i] = 98;
}

}
else
{
	if(input[i]==(97+offset2))
{
	vR[i] = 97;
}
else
{
	vR[i] = 96;
}

}

}

}

}

}
else
{
	if(input[i]>(79+offset2))
{
	if(input[i]>(87+offset2))
{
	if(input[i]>(91+offset2))
{
	if(input[i]>(93+offset2))
{
	if(input[i]==(95+offset2))
{
	vR[i] = 95;
}
else
{
	vR[i] = 94;
}

}
else
{
	if(input[i]==(93+offset2))
{
	vR[i] = 93;
}
else
{
	vR[i] = 92;
}

}

}
else
{
	if(input[i]>(89+offset2))
{
	if(input[i]==(91+offset2))
{
	vR[i] = 91;
}
else
{
	vR[i] = 90;
}

}
else
{
	if(input[i]==(89+offset2))
{
	vR[i] = 89;
}
else
{
	vR[i] = 88;
}

}

}

}
else
{
	if(input[i]>(83+offset2))
{
	if(input[i]>(85+offset2))
{
	if(input[i]==(87+offset2))
{
	vR[i] = 87;
}
else
{
	vR[i] = 86;
}

}
else
{
	if(input[i]==(85+offset2))
{
	vR[i] = 85;
}
else
{
	vR[i] = 84;
}

}

}
else
{
	if(input[i]>(81+offset2))
{
	if(input[i]==(83+offset2))
{
	vR[i] = 83;
}
else
{
	vR[i] = 82;
}

}
else
{
	if(input[i]==(81+offset2))
{
	vR[i] = 81;
}
else
{
	vR[i] = 80;
}

}

}

}

}
else
{
	if(input[i]>(71+offset2))
{
	if(input[i]>(75+offset2))
{
	if(input[i]>(77+offset2))
{
	if(input[i]==(79+offset2))
{
	vR[i] = 79;
}
else
{
	vR[i] = 78;
}

}
else
{
	if(input[i]==(77+offset2))
{
	vR[i] = 77;
}
else
{
	vR[i] = 76;
}

}

}
else
{
	if(input[i]>(73+offset2))
{
	if(input[i]==(75+offset2))
{
	vR[i] = 75;
}
else
{
	vR[i] = 74;
}

}
else
{
	if(input[i]==(73+offset2))
{
	vR[i] = 73;
}
else
{
	vR[i] = 72;
}

}

}

}
else
{
	if(input[i]>(67+offset2))
{
	if(input[i]>(69+offset2))
{
	if(input[i]==(71+offset2))
{
	vR[i] = 71;
}
else
{
	vR[i] = 70;
}

}
else
{
	if(input[i]==(69+offset2))
{
	vR[i] = 69;
}
else
{
	vR[i] = 68;
}

}

}
else
{
	if(input[i]>(65+offset2))
{
	if(input[i]==(67+offset2))
{
	vR[i] = 67;
}
else
{
	vR[i] = 66;
}

}
else
{
	if(input[i]==(65+offset2))
{
	vR[i] = 65;
}
else
{
	vR[i] = 64;
}

}

}

}

}

}

}
else
{
	if(input[i]>(31+offset2))
{
	if(input[i]>(47+offset2))
{
	if(input[i]>(55+offset2))
{
	if(input[i]>(59+offset2))
{
	if(input[i]>(61+offset2))
{
	if(input[i]==(63+offset2))
{
	vR[i] = 63;
}
else
{
	vR[i] = 62;
}

}
else
{
	if(input[i]==(61+offset2))
{
	vR[i] = 61;
}
else
{
	vR[i] = 60;
}

}

}
else
{
	if(input[i]>(57+offset2))
{
	if(input[i]==(59+offset2))
{
	vR[i] = 59;
}
else
{
	vR[i] = 58;
}

}
else
{
	if(input[i]==(57+offset2))
{
	vR[i] = 57;
}
else
{
	vR[i] = 56;
}

}

}

}
else
{
	if(input[i]>(51+offset2))
{
	if(input[i]>(53+offset2))
{
	if(input[i]==(55+offset2))
{
	vR[i] = 55;
}
else
{
	vR[i] = 54;
}

}
else
{
	if(input[i]==(53+offset2))
{
	vR[i] = 53;
}
else
{
	vR[i] = 52;
}

}

}
else
{
	if(input[i]>(49+offset2))
{
	if(input[i]==(51+offset2))
{
	vR[i] = 51;
}
else
{
	vR[i] = 50;
}

}
else
{
	if(input[i]==(49+offset2))
{
	vR[i] = 49;
}
else
{
	vR[i] = 48;
}

}

}

}

}
else
{
	if(input[i]>(39+offset2))
{
	if(input[i]>(43+offset2))
{
	if(input[i]>(45+offset2))
{
	if(input[i]==(47+offset2))
{
	vR[i] = 47;
}
else
{
	vR[i] = 46;
}

}
else
{
	if(input[i]==(45+offset2))
{
	vR[i] = 45;
}
else
{
	vR[i] = 44;
}

}

}
else
{
	if(input[i]>(41+offset2))
{
	if(input[i]==(43+offset2))
{
	vR[i] = 43;
}
else
{
	vR[i] = 42;
}

}
else
{
	if(input[i]==(41+offset2))
{
	vR[i] = 41;
}
else
{
	vR[i] = 40;
}

}

}

}
else
{
	if(input[i]>(35+offset2))
{
	if(input[i]>(37+offset2))
{
	if(input[i]==(39+offset2))
{
	vR[i] = 39;
}
else
{
	vR[i] = 38;
}

}
else
{
	if(input[i]==(37+offset2))
{
	vR[i] = 37;
}
else
{
	vR[i] = 36;
}

}

}
else
{
	if(input[i]>(33+offset2))
{
	if(input[i]==(35+offset2))
{
	vR[i] = 35;
}
else
{
	vR[i] = 34;
}

}
else
{
	if(input[i]==(33+offset2))
{
	vR[i] = 33;
}
else
{
	vR[i] = 32;
}

}

}

}

}

}
else
{
	if(input[i]>(15+offset2))
{
	if(input[i]>(23+offset2))
{
	if(input[i]>(27+offset2))
{
	if(input[i]>(29+offset2))
{
	if(input[i]==(31+offset2))
{
	vR[i] = 31;
}
else
{
	vR[i] = 30;
}

}
else
{
	if(input[i]==(29+offset2))
{
	vR[i] = 29;
}
else
{
	vR[i] = 28;
}

}

}
else
{
	if(input[i]>(25+offset2))
{
	if(input[i]==(27+offset2))
{
	vR[i] = 27;
}
else
{
	vR[i] = 26;
}

}
else
{
	if(input[i]==(25+offset2))
{
	vR[i] = 25;
}
else
{
	vR[i] = 24;
}

}

}

}
else
{
	if(input[i]>(19+offset2))
{
	if(input[i]>(21+offset2))
{
	if(input[i]==(23+offset2))
{
	vR[i] = 23;
}
else
{
	vR[i] = 22;
}

}
else
{
	if(input[i]==(21+offset2))
{
	vR[i] = 21;
}
else
{
	vR[i] = 20;
}

}

}
else
{
	if(input[i]>(17+offset2))
{
	if(input[i]==(19+offset2))
{
	vR[i] = 19;
}
else
{
	vR[i] = 18;
}

}
else
{
	if(input[i]==(17+offset2))
{
	vR[i] = 17;
}
else
{
	vR[i] = 16;
}

}

}

}

}
else
{
	if(input[i]>(7+offset2))
{
	if(input[i]>(11+offset2))
{
	if(input[i]>(13+offset2))
{
	if(input[i]==(15+offset2))
{
	vR[i] = 15;
}
else
{
	vR[i] = 14;
}

}
else
{
	if(input[i]==(13+offset2))
{
	vR[i] = 13;
}
else
{
	vR[i] = 12;
}

}

}
else
{
	if(input[i]>(9+offset2))
{
	if(input[i]==(11+offset2))
{
	vR[i] = 11;
}
else
{
	vR[i] = 10;
}

}
else
{
	if(input[i]==(9+offset2))
{
	vR[i] = 9;
}
else
{
	vR[i] = 8;
}

}

}

}
else
{
	if(input[i]>(3+offset2))
{
	if(input[i]>(5+offset2))
{
	if(input[i]==(7+offset2))
{
	vR[i] = 7;
}
else
{
	vR[i] = 6;
}

}
else
{
	if(input[i]==(5+offset2))
{
	vR[i] = 5;
}
else
{
	vR[i] = 4;
}

}

}
else
{
	if(input[i]>(1+offset2))
{
	if(input[i]==(3+offset2))
{
	vR[i] = 3;
}
else
{
	vR[i] = 2;
}

}
else
{
	if(input[i]==(1+offset2))
{
	vR[i] = 1;
}
else
{
	vR[i] = 0;
}

}

}

}

}

}

}

}

}

}

}


if(input[i]>(1023+offset3))
{
	if(input[i]>(1535+offset3))
{
	if(input[i]>(1791+offset3))
{
	if(input[i]>(1919+offset3))
{
	if(input[i]>(1983+offset3))
{
	if(input[i]>(2015+offset3))
{
	if(input[i]>(2031+offset3))
{
	if(input[i]>(2039+offset3))
{
	if(input[i]>(2043+offset3))
{
	if(input[i]>(2045+offset3))
{
	if(input[i]==(2047+offset3))
{
	vR[i] = 2047;
}
else
{
	vR[i] = 2046;
}

}
else
{
	if(input[i]==(2045+offset3))
{
	vR[i] = 2045;
}
else
{
	vR[i] = 2044;
}

}

}
else
{
	if(input[i]>(2041+offset3))
{
	if(input[i]==(2043+offset3))
{
	vR[i] = 2043;
}
else
{
	vR[i] = 2042;
}

}
else
{
	if(input[i]==(2041+offset3))
{
	vR[i] = 2041;
}
else
{
	vR[i] = 2040;
}

}

}

}
else
{
	if(input[i]>(2035+offset3))
{
	if(input[i]>(2037+offset3))
{
	if(input[i]==(2039+offset3))
{
	vR[i] = 2039;
}
else
{
	vR[i] = 2038;
}

}
else
{
	if(input[i]==(2037+offset3))
{
	vR[i] = 2037;
}
else
{
	vR[i] = 2036;
}

}

}
else
{
	if(input[i]>(2033+offset3))
{
	if(input[i]==(2035+offset3))
{
	vR[i] = 2035;
}
else
{
	vR[i] = 2034;
}

}
else
{
	if(input[i]==(2033+offset3))
{
	vR[i] = 2033;
}
else
{
	vR[i] = 2032;
}

}

}

}

}
else
{
	if(input[i]>(2023+offset3))
{
	if(input[i]>(2027+offset3))
{
	if(input[i]>(2029+offset3))
{
	if(input[i]==(2031+offset3))
{
	vR[i] = 2031;
}
else
{
	vR[i] = 2030;
}

}
else
{
	if(input[i]==(2029+offset3))
{
	vR[i] = 2029;
}
else
{
	vR[i] = 2028;
}

}

}
else
{
	if(input[i]>(2025+offset3))
{
	if(input[i]==(2027+offset3))
{
	vR[i] = 2027;
}
else
{
	vR[i] = 2026;
}

}
else
{
	if(input[i]==(2025+offset3))
{
	vR[i] = 2025;
}
else
{
	vR[i] = 2024;
}

}

}

}
else
{
	if(input[i]>(2019+offset3))
{
	if(input[i]>(2021+offset3))
{
	if(input[i]==(2023+offset3))
{
	vR[i] = 2023;
}
else
{
	vR[i] = 2022;
}

}
else
{
	if(input[i]==(2021+offset3))
{
	vR[i] = 2021;
}
else
{
	vR[i] = 2020;
}

}

}
else
{
	if(input[i]>(2017+offset3))
{
	if(input[i]==(2019+offset3))
{
	vR[i] = 2019;
}
else
{
	vR[i] = 2018;
}

}
else
{
	if(input[i]==(2017+offset3))
{
	vR[i] = 2017;
}
else
{
	vR[i] = 2016;
}

}

}

}

}

}
else
{
	if(input[i]>(1999+offset3))
{
	if(input[i]>(2007+offset3))
{
	if(input[i]>(2011+offset3))
{
	if(input[i]>(2013+offset3))
{
	if(input[i]==(2015+offset3))
{
	vR[i] = 2015;
}
else
{
	vR[i] = 2014;
}

}
else
{
	if(input[i]==(2013+offset3))
{
	vR[i] = 2013;
}
else
{
	vR[i] = 2012;
}

}

}
else
{
	if(input[i]>(2009+offset3))
{
	if(input[i]==(2011+offset3))
{
	vR[i] = 2011;
}
else
{
	vR[i] = 2010;
}

}
else
{
	if(input[i]==(2009+offset3))
{
	vR[i] = 2009;
}
else
{
	vR[i] = 2008;
}

}

}

}
else
{
	if(input[i]>(2003+offset3))
{
	if(input[i]>(2005+offset3))
{
	if(input[i]==(2007+offset3))
{
	vR[i] = 2007;
}
else
{
	vR[i] = 2006;
}

}
else
{
	if(input[i]==(2005+offset3))
{
	vR[i] = 2005;
}
else
{
	vR[i] = 2004;
}

}

}
else
{
	if(input[i]>(2001+offset3))
{
	if(input[i]==(2003+offset3))
{
	vR[i] = 2003;
}
else
{
	vR[i] = 2002;
}

}
else
{
	if(input[i]==(2001+offset3))
{
	vR[i] = 2001;
}
else
{
	vR[i] = 2000;
}

}

}

}

}
else
{
	if(input[i]>(1991+offset3))
{
	if(input[i]>(1995+offset3))
{
	if(input[i]>(1997+offset3))
{
	if(input[i]==(1999+offset3))
{
	vR[i] = 1999;
}
else
{
	vR[i] = 1998;
}

}
else
{
	if(input[i]==(1997+offset3))
{
	vR[i] = 1997;
}
else
{
	vR[i] = 1996;
}

}

}
else
{
	if(input[i]>(1993+offset3))
{
	if(input[i]==(1995+offset3))
{
	vR[i] = 1995;
}
else
{
	vR[i] = 1994;
}

}
else
{
	if(input[i]==(1993+offset3))
{
	vR[i] = 1993;
}
else
{
	vR[i] = 1992;
}

}

}

}
else
{
	if(input[i]>(1987+offset3))
{
	if(input[i]>(1989+offset3))
{
	if(input[i]==(1991+offset3))
{
	vR[i] = 1991;
}
else
{
	vR[i] = 1990;
}

}
else
{
	if(input[i]==(1989+offset3))
{
	vR[i] = 1989;
}
else
{
	vR[i] = 1988;
}

}

}
else
{
	if(input[i]>(1985+offset3))
{
	if(input[i]==(1987+offset3))
{
	vR[i] = 1987;
}
else
{
	vR[i] = 1986;
}

}
else
{
	if(input[i]==(1985+offset3))
{
	vR[i] = 1985;
}
else
{
	vR[i] = 1984;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1951+offset3))
{
	if(input[i]>(1967+offset3))
{
	if(input[i]>(1975+offset3))
{
	if(input[i]>(1979+offset3))
{
	if(input[i]>(1981+offset3))
{
	if(input[i]==(1983+offset3))
{
	vR[i] = 1983;
}
else
{
	vR[i] = 1982;
}

}
else
{
	if(input[i]==(1981+offset3))
{
	vR[i] = 1981;
}
else
{
	vR[i] = 1980;
}

}

}
else
{
	if(input[i]>(1977+offset3))
{
	if(input[i]==(1979+offset3))
{
	vR[i] = 1979;
}
else
{
	vR[i] = 1978;
}

}
else
{
	if(input[i]==(1977+offset3))
{
	vR[i] = 1977;
}
else
{
	vR[i] = 1976;
}

}

}

}
else
{
	if(input[i]>(1971+offset3))
{
	if(input[i]>(1973+offset3))
{
	if(input[i]==(1975+offset3))
{
	vR[i] = 1975;
}
else
{
	vR[i] = 1974;
}

}
else
{
	if(input[i]==(1973+offset3))
{
	vR[i] = 1973;
}
else
{
	vR[i] = 1972;
}

}

}
else
{
	if(input[i]>(1969+offset3))
{
	if(input[i]==(1971+offset3))
{
	vR[i] = 1971;
}
else
{
	vR[i] = 1970;
}

}
else
{
	if(input[i]==(1969+offset3))
{
	vR[i] = 1969;
}
else
{
	vR[i] = 1968;
}

}

}

}

}
else
{
	if(input[i]>(1959+offset3))
{
	if(input[i]>(1963+offset3))
{
	if(input[i]>(1965+offset3))
{
	if(input[i]==(1967+offset3))
{
	vR[i] = 1967;
}
else
{
	vR[i] = 1966;
}

}
else
{
	if(input[i]==(1965+offset3))
{
	vR[i] = 1965;
}
else
{
	vR[i] = 1964;
}

}

}
else
{
	if(input[i]>(1961+offset3))
{
	if(input[i]==(1963+offset3))
{
	vR[i] = 1963;
}
else
{
	vR[i] = 1962;
}

}
else
{
	if(input[i]==(1961+offset3))
{
	vR[i] = 1961;
}
else
{
	vR[i] = 1960;
}

}

}

}
else
{
	if(input[i]>(1955+offset3))
{
	if(input[i]>(1957+offset3))
{
	if(input[i]==(1959+offset3))
{
	vR[i] = 1959;
}
else
{
	vR[i] = 1958;
}

}
else
{
	if(input[i]==(1957+offset3))
{
	vR[i] = 1957;
}
else
{
	vR[i] = 1956;
}

}

}
else
{
	if(input[i]>(1953+offset3))
{
	if(input[i]==(1955+offset3))
{
	vR[i] = 1955;
}
else
{
	vR[i] = 1954;
}

}
else
{
	if(input[i]==(1953+offset3))
{
	vR[i] = 1953;
}
else
{
	vR[i] = 1952;
}

}

}

}

}

}
else
{
	if(input[i]>(1935+offset3))
{
	if(input[i]>(1943+offset3))
{
	if(input[i]>(1947+offset3))
{
	if(input[i]>(1949+offset3))
{
	if(input[i]==(1951+offset3))
{
	vR[i] = 1951;
}
else
{
	vR[i] = 1950;
}

}
else
{
	if(input[i]==(1949+offset3))
{
	vR[i] = 1949;
}
else
{
	vR[i] = 1948;
}

}

}
else
{
	if(input[i]>(1945+offset3))
{
	if(input[i]==(1947+offset3))
{
	vR[i] = 1947;
}
else
{
	vR[i] = 1946;
}

}
else
{
	if(input[i]==(1945+offset3))
{
	vR[i] = 1945;
}
else
{
	vR[i] = 1944;
}

}

}

}
else
{
	if(input[i]>(1939+offset3))
{
	if(input[i]>(1941+offset3))
{
	if(input[i]==(1943+offset3))
{
	vR[i] = 1943;
}
else
{
	vR[i] = 1942;
}

}
else
{
	if(input[i]==(1941+offset3))
{
	vR[i] = 1941;
}
else
{
	vR[i] = 1940;
}

}

}
else
{
	if(input[i]>(1937+offset3))
{
	if(input[i]==(1939+offset3))
{
	vR[i] = 1939;
}
else
{
	vR[i] = 1938;
}

}
else
{
	if(input[i]==(1937+offset3))
{
	vR[i] = 1937;
}
else
{
	vR[i] = 1936;
}

}

}

}

}
else
{
	if(input[i]>(1927+offset3))
{
	if(input[i]>(1931+offset3))
{
	if(input[i]>(1933+offset3))
{
	if(input[i]==(1935+offset3))
{
	vR[i] = 1935;
}
else
{
	vR[i] = 1934;
}

}
else
{
	if(input[i]==(1933+offset3))
{
	vR[i] = 1933;
}
else
{
	vR[i] = 1932;
}

}

}
else
{
	if(input[i]>(1929+offset3))
{
	if(input[i]==(1931+offset3))
{
	vR[i] = 1931;
}
else
{
	vR[i] = 1930;
}

}
else
{
	if(input[i]==(1929+offset3))
{
	vR[i] = 1929;
}
else
{
	vR[i] = 1928;
}

}

}

}
else
{
	if(input[i]>(1923+offset3))
{
	if(input[i]>(1925+offset3))
{
	if(input[i]==(1927+offset3))
{
	vR[i] = 1927;
}
else
{
	vR[i] = 1926;
}

}
else
{
	if(input[i]==(1925+offset3))
{
	vR[i] = 1925;
}
else
{
	vR[i] = 1924;
}

}

}
else
{
	if(input[i]>(1921+offset3))
{
	if(input[i]==(1923+offset3))
{
	vR[i] = 1923;
}
else
{
	vR[i] = 1922;
}

}
else
{
	if(input[i]==(1921+offset3))
{
	vR[i] = 1921;
}
else
{
	vR[i] = 1920;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(1855+offset3))
{
	if(input[i]>(1887+offset3))
{
	if(input[i]>(1903+offset3))
{
	if(input[i]>(1911+offset3))
{
	if(input[i]>(1915+offset3))
{
	if(input[i]>(1917+offset3))
{
	if(input[i]==(1919+offset3))
{
	vR[i] = 1919;
}
else
{
	vR[i] = 1918;
}

}
else
{
	if(input[i]==(1917+offset3))
{
	vR[i] = 1917;
}
else
{
	vR[i] = 1916;
}

}

}
else
{
	if(input[i]>(1913+offset3))
{
	if(input[i]==(1915+offset3))
{
	vR[i] = 1915;
}
else
{
	vR[i] = 1914;
}

}
else
{
	if(input[i]==(1913+offset3))
{
	vR[i] = 1913;
}
else
{
	vR[i] = 1912;
}

}

}

}
else
{
	if(input[i]>(1907+offset3))
{
	if(input[i]>(1909+offset3))
{
	if(input[i]==(1911+offset3))
{
	vR[i] = 1911;
}
else
{
	vR[i] = 1910;
}

}
else
{
	if(input[i]==(1909+offset3))
{
	vR[i] = 1909;
}
else
{
	vR[i] = 1908;
}

}

}
else
{
	if(input[i]>(1905+offset3))
{
	if(input[i]==(1907+offset3))
{
	vR[i] = 1907;
}
else
{
	vR[i] = 1906;
}

}
else
{
	if(input[i]==(1905+offset3))
{
	vR[i] = 1905;
}
else
{
	vR[i] = 1904;
}

}

}

}

}
else
{
	if(input[i]>(1895+offset3))
{
	if(input[i]>(1899+offset3))
{
	if(input[i]>(1901+offset3))
{
	if(input[i]==(1903+offset3))
{
	vR[i] = 1903;
}
else
{
	vR[i] = 1902;
}

}
else
{
	if(input[i]==(1901+offset3))
{
	vR[i] = 1901;
}
else
{
	vR[i] = 1900;
}

}

}
else
{
	if(input[i]>(1897+offset3))
{
	if(input[i]==(1899+offset3))
{
	vR[i] = 1899;
}
else
{
	vR[i] = 1898;
}

}
else
{
	if(input[i]==(1897+offset3))
{
	vR[i] = 1897;
}
else
{
	vR[i] = 1896;
}

}

}

}
else
{
	if(input[i]>(1891+offset3))
{
	if(input[i]>(1893+offset3))
{
	if(input[i]==(1895+offset3))
{
	vR[i] = 1895;
}
else
{
	vR[i] = 1894;
}

}
else
{
	if(input[i]==(1893+offset3))
{
	vR[i] = 1893;
}
else
{
	vR[i] = 1892;
}

}

}
else
{
	if(input[i]>(1889+offset3))
{
	if(input[i]==(1891+offset3))
{
	vR[i] = 1891;
}
else
{
	vR[i] = 1890;
}

}
else
{
	if(input[i]==(1889+offset3))
{
	vR[i] = 1889;
}
else
{
	vR[i] = 1888;
}

}

}

}

}

}
else
{
	if(input[i]>(1871+offset3))
{
	if(input[i]>(1879+offset3))
{
	if(input[i]>(1883+offset3))
{
	if(input[i]>(1885+offset3))
{
	if(input[i]==(1887+offset3))
{
	vR[i] = 1887;
}
else
{
	vR[i] = 1886;
}

}
else
{
	if(input[i]==(1885+offset3))
{
	vR[i] = 1885;
}
else
{
	vR[i] = 1884;
}

}

}
else
{
	if(input[i]>(1881+offset3))
{
	if(input[i]==(1883+offset3))
{
	vR[i] = 1883;
}
else
{
	vR[i] = 1882;
}

}
else
{
	if(input[i]==(1881+offset3))
{
	vR[i] = 1881;
}
else
{
	vR[i] = 1880;
}

}

}

}
else
{
	if(input[i]>(1875+offset3))
{
	if(input[i]>(1877+offset3))
{
	if(input[i]==(1879+offset3))
{
	vR[i] = 1879;
}
else
{
	vR[i] = 1878;
}

}
else
{
	if(input[i]==(1877+offset3))
{
	vR[i] = 1877;
}
else
{
	vR[i] = 1876;
}

}

}
else
{
	if(input[i]>(1873+offset3))
{
	if(input[i]==(1875+offset3))
{
	vR[i] = 1875;
}
else
{
	vR[i] = 1874;
}

}
else
{
	if(input[i]==(1873+offset3))
{
	vR[i] = 1873;
}
else
{
	vR[i] = 1872;
}

}

}

}

}
else
{
	if(input[i]>(1863+offset3))
{
	if(input[i]>(1867+offset3))
{
	if(input[i]>(1869+offset3))
{
	if(input[i]==(1871+offset3))
{
	vR[i] = 1871;
}
else
{
	vR[i] = 1870;
}

}
else
{
	if(input[i]==(1869+offset3))
{
	vR[i] = 1869;
}
else
{
	vR[i] = 1868;
}

}

}
else
{
	if(input[i]>(1865+offset3))
{
	if(input[i]==(1867+offset3))
{
	vR[i] = 1867;
}
else
{
	vR[i] = 1866;
}

}
else
{
	if(input[i]==(1865+offset3))
{
	vR[i] = 1865;
}
else
{
	vR[i] = 1864;
}

}

}

}
else
{
	if(input[i]>(1859+offset3))
{
	if(input[i]>(1861+offset3))
{
	if(input[i]==(1863+offset3))
{
	vR[i] = 1863;
}
else
{
	vR[i] = 1862;
}

}
else
{
	if(input[i]==(1861+offset3))
{
	vR[i] = 1861;
}
else
{
	vR[i] = 1860;
}

}

}
else
{
	if(input[i]>(1857+offset3))
{
	if(input[i]==(1859+offset3))
{
	vR[i] = 1859;
}
else
{
	vR[i] = 1858;
}

}
else
{
	if(input[i]==(1857+offset3))
{
	vR[i] = 1857;
}
else
{
	vR[i] = 1856;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1823+offset3))
{
	if(input[i]>(1839+offset3))
{
	if(input[i]>(1847+offset3))
{
	if(input[i]>(1851+offset3))
{
	if(input[i]>(1853+offset3))
{
	if(input[i]==(1855+offset3))
{
	vR[i] = 1855;
}
else
{
	vR[i] = 1854;
}

}
else
{
	if(input[i]==(1853+offset3))
{
	vR[i] = 1853;
}
else
{
	vR[i] = 1852;
}

}

}
else
{
	if(input[i]>(1849+offset3))
{
	if(input[i]==(1851+offset3))
{
	vR[i] = 1851;
}
else
{
	vR[i] = 1850;
}

}
else
{
	if(input[i]==(1849+offset3))
{
	vR[i] = 1849;
}
else
{
	vR[i] = 1848;
}

}

}

}
else
{
	if(input[i]>(1843+offset3))
{
	if(input[i]>(1845+offset3))
{
	if(input[i]==(1847+offset3))
{
	vR[i] = 1847;
}
else
{
	vR[i] = 1846;
}

}
else
{
	if(input[i]==(1845+offset3))
{
	vR[i] = 1845;
}
else
{
	vR[i] = 1844;
}

}

}
else
{
	if(input[i]>(1841+offset3))
{
	if(input[i]==(1843+offset3))
{
	vR[i] = 1843;
}
else
{
	vR[i] = 1842;
}

}
else
{
	if(input[i]==(1841+offset3))
{
	vR[i] = 1841;
}
else
{
	vR[i] = 1840;
}

}

}

}

}
else
{
	if(input[i]>(1831+offset3))
{
	if(input[i]>(1835+offset3))
{
	if(input[i]>(1837+offset3))
{
	if(input[i]==(1839+offset3))
{
	vR[i] = 1839;
}
else
{
	vR[i] = 1838;
}

}
else
{
	if(input[i]==(1837+offset3))
{
	vR[i] = 1837;
}
else
{
	vR[i] = 1836;
}

}

}
else
{
	if(input[i]>(1833+offset3))
{
	if(input[i]==(1835+offset3))
{
	vR[i] = 1835;
}
else
{
	vR[i] = 1834;
}

}
else
{
	if(input[i]==(1833+offset3))
{
	vR[i] = 1833;
}
else
{
	vR[i] = 1832;
}

}

}

}
else
{
	if(input[i]>(1827+offset3))
{
	if(input[i]>(1829+offset3))
{
	if(input[i]==(1831+offset3))
{
	vR[i] = 1831;
}
else
{
	vR[i] = 1830;
}

}
else
{
	if(input[i]==(1829+offset3))
{
	vR[i] = 1829;
}
else
{
	vR[i] = 1828;
}

}

}
else
{
	if(input[i]>(1825+offset3))
{
	if(input[i]==(1827+offset3))
{
	vR[i] = 1827;
}
else
{
	vR[i] = 1826;
}

}
else
{
	if(input[i]==(1825+offset3))
{
	vR[i] = 1825;
}
else
{
	vR[i] = 1824;
}

}

}

}

}

}
else
{
	if(input[i]>(1807+offset3))
{
	if(input[i]>(1815+offset3))
{
	if(input[i]>(1819+offset3))
{
	if(input[i]>(1821+offset3))
{
	if(input[i]==(1823+offset3))
{
	vR[i] = 1823;
}
else
{
	vR[i] = 1822;
}

}
else
{
	if(input[i]==(1821+offset3))
{
	vR[i] = 1821;
}
else
{
	vR[i] = 1820;
}

}

}
else
{
	if(input[i]>(1817+offset3))
{
	if(input[i]==(1819+offset3))
{
	vR[i] = 1819;
}
else
{
	vR[i] = 1818;
}

}
else
{
	if(input[i]==(1817+offset3))
{
	vR[i] = 1817;
}
else
{
	vR[i] = 1816;
}

}

}

}
else
{
	if(input[i]>(1811+offset3))
{
	if(input[i]>(1813+offset3))
{
	if(input[i]==(1815+offset3))
{
	vR[i] = 1815;
}
else
{
	vR[i] = 1814;
}

}
else
{
	if(input[i]==(1813+offset3))
{
	vR[i] = 1813;
}
else
{
	vR[i] = 1812;
}

}

}
else
{
	if(input[i]>(1809+offset3))
{
	if(input[i]==(1811+offset3))
{
	vR[i] = 1811;
}
else
{
	vR[i] = 1810;
}

}
else
{
	if(input[i]==(1809+offset3))
{
	vR[i] = 1809;
}
else
{
	vR[i] = 1808;
}

}

}

}

}
else
{
	if(input[i]>(1799+offset3))
{
	if(input[i]>(1803+offset3))
{
	if(input[i]>(1805+offset3))
{
	if(input[i]==(1807+offset3))
{
	vR[i] = 1807;
}
else
{
	vR[i] = 1806;
}

}
else
{
	if(input[i]==(1805+offset3))
{
	vR[i] = 1805;
}
else
{
	vR[i] = 1804;
}

}

}
else
{
	if(input[i]>(1801+offset3))
{
	if(input[i]==(1803+offset3))
{
	vR[i] = 1803;
}
else
{
	vR[i] = 1802;
}

}
else
{
	if(input[i]==(1801+offset3))
{
	vR[i] = 1801;
}
else
{
	vR[i] = 1800;
}

}

}

}
else
{
	if(input[i]>(1795+offset3))
{
	if(input[i]>(1797+offset3))
{
	if(input[i]==(1799+offset3))
{
	vR[i] = 1799;
}
else
{
	vR[i] = 1798;
}

}
else
{
	if(input[i]==(1797+offset3))
{
	vR[i] = 1797;
}
else
{
	vR[i] = 1796;
}

}

}
else
{
	if(input[i]>(1793+offset3))
{
	if(input[i]==(1795+offset3))
{
	vR[i] = 1795;
}
else
{
	vR[i] = 1794;
}

}
else
{
	if(input[i]==(1793+offset3))
{
	vR[i] = 1793;
}
else
{
	vR[i] = 1792;
}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(1663+offset3))
{
	if(input[i]>(1727+offset3))
{
	if(input[i]>(1759+offset3))
{
	if(input[i]>(1775+offset3))
{
	if(input[i]>(1783+offset3))
{
	if(input[i]>(1787+offset3))
{
	if(input[i]>(1789+offset3))
{
	if(input[i]==(1791+offset3))
{
	vR[i] = 1791;
}
else
{
	vR[i] = 1790;
}

}
else
{
	if(input[i]==(1789+offset3))
{
	vR[i] = 1789;
}
else
{
	vR[i] = 1788;
}

}

}
else
{
	if(input[i]>(1785+offset3))
{
	if(input[i]==(1787+offset3))
{
	vR[i] = 1787;
}
else
{
	vR[i] = 1786;
}

}
else
{
	if(input[i]==(1785+offset3))
{
	vR[i] = 1785;
}
else
{
	vR[i] = 1784;
}

}

}

}
else
{
	if(input[i]>(1779+offset3))
{
	if(input[i]>(1781+offset3))
{
	if(input[i]==(1783+offset3))
{
	vR[i] = 1783;
}
else
{
	vR[i] = 1782;
}

}
else
{
	if(input[i]==(1781+offset3))
{
	vR[i] = 1781;
}
else
{
	vR[i] = 1780;
}

}

}
else
{
	if(input[i]>(1777+offset3))
{
	if(input[i]==(1779+offset3))
{
	vR[i] = 1779;
}
else
{
	vR[i] = 1778;
}

}
else
{
	if(input[i]==(1777+offset3))
{
	vR[i] = 1777;
}
else
{
	vR[i] = 1776;
}

}

}

}

}
else
{
	if(input[i]>(1767+offset3))
{
	if(input[i]>(1771+offset3))
{
	if(input[i]>(1773+offset3))
{
	if(input[i]==(1775+offset3))
{
	vR[i] = 1775;
}
else
{
	vR[i] = 1774;
}

}
else
{
	if(input[i]==(1773+offset3))
{
	vR[i] = 1773;
}
else
{
	vR[i] = 1772;
}

}

}
else
{
	if(input[i]>(1769+offset3))
{
	if(input[i]==(1771+offset3))
{
	vR[i] = 1771;
}
else
{
	vR[i] = 1770;
}

}
else
{
	if(input[i]==(1769+offset3))
{
	vR[i] = 1769;
}
else
{
	vR[i] = 1768;
}

}

}

}
else
{
	if(input[i]>(1763+offset3))
{
	if(input[i]>(1765+offset3))
{
	if(input[i]==(1767+offset3))
{
	vR[i] = 1767;
}
else
{
	vR[i] = 1766;
}

}
else
{
	if(input[i]==(1765+offset3))
{
	vR[i] = 1765;
}
else
{
	vR[i] = 1764;
}

}

}
else
{
	if(input[i]>(1761+offset3))
{
	if(input[i]==(1763+offset3))
{
	vR[i] = 1763;
}
else
{
	vR[i] = 1762;
}

}
else
{
	if(input[i]==(1761+offset3))
{
	vR[i] = 1761;
}
else
{
	vR[i] = 1760;
}

}

}

}

}

}
else
{
	if(input[i]>(1743+offset3))
{
	if(input[i]>(1751+offset3))
{
	if(input[i]>(1755+offset3))
{
	if(input[i]>(1757+offset3))
{
	if(input[i]==(1759+offset3))
{
	vR[i] = 1759;
}
else
{
	vR[i] = 1758;
}

}
else
{
	if(input[i]==(1757+offset3))
{
	vR[i] = 1757;
}
else
{
	vR[i] = 1756;
}

}

}
else
{
	if(input[i]>(1753+offset3))
{
	if(input[i]==(1755+offset3))
{
	vR[i] = 1755;
}
else
{
	vR[i] = 1754;
}

}
else
{
	if(input[i]==(1753+offset3))
{
	vR[i] = 1753;
}
else
{
	vR[i] = 1752;
}

}

}

}
else
{
	if(input[i]>(1747+offset3))
{
	if(input[i]>(1749+offset3))
{
	if(input[i]==(1751+offset3))
{
	vR[i] = 1751;
}
else
{
	vR[i] = 1750;
}

}
else
{
	if(input[i]==(1749+offset3))
{
	vR[i] = 1749;
}
else
{
	vR[i] = 1748;
}

}

}
else
{
	if(input[i]>(1745+offset3))
{
	if(input[i]==(1747+offset3))
{
	vR[i] = 1747;
}
else
{
	vR[i] = 1746;
}

}
else
{
	if(input[i]==(1745+offset3))
{
	vR[i] = 1745;
}
else
{
	vR[i] = 1744;
}

}

}

}

}
else
{
	if(input[i]>(1735+offset3))
{
	if(input[i]>(1739+offset3))
{
	if(input[i]>(1741+offset3))
{
	if(input[i]==(1743+offset3))
{
	vR[i] = 1743;
}
else
{
	vR[i] = 1742;
}

}
else
{
	if(input[i]==(1741+offset3))
{
	vR[i] = 1741;
}
else
{
	vR[i] = 1740;
}

}

}
else
{
	if(input[i]>(1737+offset3))
{
	if(input[i]==(1739+offset3))
{
	vR[i] = 1739;
}
else
{
	vR[i] = 1738;
}

}
else
{
	if(input[i]==(1737+offset3))
{
	vR[i] = 1737;
}
else
{
	vR[i] = 1736;
}

}

}

}
else
{
	if(input[i]>(1731+offset3))
{
	if(input[i]>(1733+offset3))
{
	if(input[i]==(1735+offset3))
{
	vR[i] = 1735;
}
else
{
	vR[i] = 1734;
}

}
else
{
	if(input[i]==(1733+offset3))
{
	vR[i] = 1733;
}
else
{
	vR[i] = 1732;
}

}

}
else
{
	if(input[i]>(1729+offset3))
{
	if(input[i]==(1731+offset3))
{
	vR[i] = 1731;
}
else
{
	vR[i] = 1730;
}

}
else
{
	if(input[i]==(1729+offset3))
{
	vR[i] = 1729;
}
else
{
	vR[i] = 1728;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1695+offset3))
{
	if(input[i]>(1711+offset3))
{
	if(input[i]>(1719+offset3))
{
	if(input[i]>(1723+offset3))
{
	if(input[i]>(1725+offset3))
{
	if(input[i]==(1727+offset3))
{
	vR[i] = 1727;
}
else
{
	vR[i] = 1726;
}

}
else
{
	if(input[i]==(1725+offset3))
{
	vR[i] = 1725;
}
else
{
	vR[i] = 1724;
}

}

}
else
{
	if(input[i]>(1721+offset3))
{
	if(input[i]==(1723+offset3))
{
	vR[i] = 1723;
}
else
{
	vR[i] = 1722;
}

}
else
{
	if(input[i]==(1721+offset3))
{
	vR[i] = 1721;
}
else
{
	vR[i] = 1720;
}

}

}

}
else
{
	if(input[i]>(1715+offset3))
{
	if(input[i]>(1717+offset3))
{
	if(input[i]==(1719+offset3))
{
	vR[i] = 1719;
}
else
{
	vR[i] = 1718;
}

}
else
{
	if(input[i]==(1717+offset3))
{
	vR[i] = 1717;
}
else
{
	vR[i] = 1716;
}

}

}
else
{
	if(input[i]>(1713+offset3))
{
	if(input[i]==(1715+offset3))
{
	vR[i] = 1715;
}
else
{
	vR[i] = 1714;
}

}
else
{
	if(input[i]==(1713+offset3))
{
	vR[i] = 1713;
}
else
{
	vR[i] = 1712;
}

}

}

}

}
else
{
	if(input[i]>(1703+offset3))
{
	if(input[i]>(1707+offset3))
{
	if(input[i]>(1709+offset3))
{
	if(input[i]==(1711+offset3))
{
	vR[i] = 1711;
}
else
{
	vR[i] = 1710;
}

}
else
{
	if(input[i]==(1709+offset3))
{
	vR[i] = 1709;
}
else
{
	vR[i] = 1708;
}

}

}
else
{
	if(input[i]>(1705+offset3))
{
	if(input[i]==(1707+offset3))
{
	vR[i] = 1707;
}
else
{
	vR[i] = 1706;
}

}
else
{
	if(input[i]==(1705+offset3))
{
	vR[i] = 1705;
}
else
{
	vR[i] = 1704;
}

}

}

}
else
{
	if(input[i]>(1699+offset3))
{
	if(input[i]>(1701+offset3))
{
	if(input[i]==(1703+offset3))
{
	vR[i] = 1703;
}
else
{
	vR[i] = 1702;
}

}
else
{
	if(input[i]==(1701+offset3))
{
	vR[i] = 1701;
}
else
{
	vR[i] = 1700;
}

}

}
else
{
	if(input[i]>(1697+offset3))
{
	if(input[i]==(1699+offset3))
{
	vR[i] = 1699;
}
else
{
	vR[i] = 1698;
}

}
else
{
	if(input[i]==(1697+offset3))
{
	vR[i] = 1697;
}
else
{
	vR[i] = 1696;
}

}

}

}

}

}
else
{
	if(input[i]>(1679+offset3))
{
	if(input[i]>(1687+offset3))
{
	if(input[i]>(1691+offset3))
{
	if(input[i]>(1693+offset3))
{
	if(input[i]==(1695+offset3))
{
	vR[i] = 1695;
}
else
{
	vR[i] = 1694;
}

}
else
{
	if(input[i]==(1693+offset3))
{
	vR[i] = 1693;
}
else
{
	vR[i] = 1692;
}

}

}
else
{
	if(input[i]>(1689+offset3))
{
	if(input[i]==(1691+offset3))
{
	vR[i] = 1691;
}
else
{
	vR[i] = 1690;
}

}
else
{
	if(input[i]==(1689+offset3))
{
	vR[i] = 1689;
}
else
{
	vR[i] = 1688;
}

}

}

}
else
{
	if(input[i]>(1683+offset3))
{
	if(input[i]>(1685+offset3))
{
	if(input[i]==(1687+offset3))
{
	vR[i] = 1687;
}
else
{
	vR[i] = 1686;
}

}
else
{
	if(input[i]==(1685+offset3))
{
	vR[i] = 1685;
}
else
{
	vR[i] = 1684;
}

}

}
else
{
	if(input[i]>(1681+offset3))
{
	if(input[i]==(1683+offset3))
{
	vR[i] = 1683;
}
else
{
	vR[i] = 1682;
}

}
else
{
	if(input[i]==(1681+offset3))
{
	vR[i] = 1681;
}
else
{
	vR[i] = 1680;
}

}

}

}

}
else
{
	if(input[i]>(1671+offset3))
{
	if(input[i]>(1675+offset3))
{
	if(input[i]>(1677+offset3))
{
	if(input[i]==(1679+offset3))
{
	vR[i] = 1679;
}
else
{
	vR[i] = 1678;
}

}
else
{
	if(input[i]==(1677+offset3))
{
	vR[i] = 1677;
}
else
{
	vR[i] = 1676;
}

}

}
else
{
	if(input[i]>(1673+offset3))
{
	if(input[i]==(1675+offset3))
{
	vR[i] = 1675;
}
else
{
	vR[i] = 1674;
}

}
else
{
	if(input[i]==(1673+offset3))
{
	vR[i] = 1673;
}
else
{
	vR[i] = 1672;
}

}

}

}
else
{
	if(input[i]>(1667+offset3))
{
	if(input[i]>(1669+offset3))
{
	if(input[i]==(1671+offset3))
{
	vR[i] = 1671;
}
else
{
	vR[i] = 1670;
}

}
else
{
	if(input[i]==(1669+offset3))
{
	vR[i] = 1669;
}
else
{
	vR[i] = 1668;
}

}

}
else
{
	if(input[i]>(1665+offset3))
{
	if(input[i]==(1667+offset3))
{
	vR[i] = 1667;
}
else
{
	vR[i] = 1666;
}

}
else
{
	if(input[i]==(1665+offset3))
{
	vR[i] = 1665;
}
else
{
	vR[i] = 1664;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(1599+offset3))
{
	if(input[i]>(1631+offset3))
{
	if(input[i]>(1647+offset3))
{
	if(input[i]>(1655+offset3))
{
	if(input[i]>(1659+offset3))
{
	if(input[i]>(1661+offset3))
{
	if(input[i]==(1663+offset3))
{
	vR[i] = 1663;
}
else
{
	vR[i] = 1662;
}

}
else
{
	if(input[i]==(1661+offset3))
{
	vR[i] = 1661;
}
else
{
	vR[i] = 1660;
}

}

}
else
{
	if(input[i]>(1657+offset3))
{
	if(input[i]==(1659+offset3))
{
	vR[i] = 1659;
}
else
{
	vR[i] = 1658;
}

}
else
{
	if(input[i]==(1657+offset3))
{
	vR[i] = 1657;
}
else
{
	vR[i] = 1656;
}

}

}

}
else
{
	if(input[i]>(1651+offset3))
{
	if(input[i]>(1653+offset3))
{
	if(input[i]==(1655+offset3))
{
	vR[i] = 1655;
}
else
{
	vR[i] = 1654;
}

}
else
{
	if(input[i]==(1653+offset3))
{
	vR[i] = 1653;
}
else
{
	vR[i] = 1652;
}

}

}
else
{
	if(input[i]>(1649+offset3))
{
	if(input[i]==(1651+offset3))
{
	vR[i] = 1651;
}
else
{
	vR[i] = 1650;
}

}
else
{
	if(input[i]==(1649+offset3))
{
	vR[i] = 1649;
}
else
{
	vR[i] = 1648;
}

}

}

}

}
else
{
	if(input[i]>(1639+offset3))
{
	if(input[i]>(1643+offset3))
{
	if(input[i]>(1645+offset3))
{
	if(input[i]==(1647+offset3))
{
	vR[i] = 1647;
}
else
{
	vR[i] = 1646;
}

}
else
{
	if(input[i]==(1645+offset3))
{
	vR[i] = 1645;
}
else
{
	vR[i] = 1644;
}

}

}
else
{
	if(input[i]>(1641+offset3))
{
	if(input[i]==(1643+offset3))
{
	vR[i] = 1643;
}
else
{
	vR[i] = 1642;
}

}
else
{
	if(input[i]==(1641+offset3))
{
	vR[i] = 1641;
}
else
{
	vR[i] = 1640;
}

}

}

}
else
{
	if(input[i]>(1635+offset3))
{
	if(input[i]>(1637+offset3))
{
	if(input[i]==(1639+offset3))
{
	vR[i] = 1639;
}
else
{
	vR[i] = 1638;
}

}
else
{
	if(input[i]==(1637+offset3))
{
	vR[i] = 1637;
}
else
{
	vR[i] = 1636;
}

}

}
else
{
	if(input[i]>(1633+offset3))
{
	if(input[i]==(1635+offset3))
{
	vR[i] = 1635;
}
else
{
	vR[i] = 1634;
}

}
else
{
	if(input[i]==(1633+offset3))
{
	vR[i] = 1633;
}
else
{
	vR[i] = 1632;
}

}

}

}

}

}
else
{
	if(input[i]>(1615+offset3))
{
	if(input[i]>(1623+offset3))
{
	if(input[i]>(1627+offset3))
{
	if(input[i]>(1629+offset3))
{
	if(input[i]==(1631+offset3))
{
	vR[i] = 1631;
}
else
{
	vR[i] = 1630;
}

}
else
{
	if(input[i]==(1629+offset3))
{
	vR[i] = 1629;
}
else
{
	vR[i] = 1628;
}

}

}
else
{
	if(input[i]>(1625+offset3))
{
	if(input[i]==(1627+offset3))
{
	vR[i] = 1627;
}
else
{
	vR[i] = 1626;
}

}
else
{
	if(input[i]==(1625+offset3))
{
	vR[i] = 1625;
}
else
{
	vR[i] = 1624;
}

}

}

}
else
{
	if(input[i]>(1619+offset3))
{
	if(input[i]>(1621+offset3))
{
	if(input[i]==(1623+offset3))
{
	vR[i] = 1623;
}
else
{
	vR[i] = 1622;
}

}
else
{
	if(input[i]==(1621+offset3))
{
	vR[i] = 1621;
}
else
{
	vR[i] = 1620;
}

}

}
else
{
	if(input[i]>(1617+offset3))
{
	if(input[i]==(1619+offset3))
{
	vR[i] = 1619;
}
else
{
	vR[i] = 1618;
}

}
else
{
	if(input[i]==(1617+offset3))
{
	vR[i] = 1617;
}
else
{
	vR[i] = 1616;
}

}

}

}

}
else
{
	if(input[i]>(1607+offset3))
{
	if(input[i]>(1611+offset3))
{
	if(input[i]>(1613+offset3))
{
	if(input[i]==(1615+offset3))
{
	vR[i] = 1615;
}
else
{
	vR[i] = 1614;
}

}
else
{
	if(input[i]==(1613+offset3))
{
	vR[i] = 1613;
}
else
{
	vR[i] = 1612;
}

}

}
else
{
	if(input[i]>(1609+offset3))
{
	if(input[i]==(1611+offset3))
{
	vR[i] = 1611;
}
else
{
	vR[i] = 1610;
}

}
else
{
	if(input[i]==(1609+offset3))
{
	vR[i] = 1609;
}
else
{
	vR[i] = 1608;
}

}

}

}
else
{
	if(input[i]>(1603+offset3))
{
	if(input[i]>(1605+offset3))
{
	if(input[i]==(1607+offset3))
{
	vR[i] = 1607;
}
else
{
	vR[i] = 1606;
}

}
else
{
	if(input[i]==(1605+offset3))
{
	vR[i] = 1605;
}
else
{
	vR[i] = 1604;
}

}

}
else
{
	if(input[i]>(1601+offset3))
{
	if(input[i]==(1603+offset3))
{
	vR[i] = 1603;
}
else
{
	vR[i] = 1602;
}

}
else
{
	if(input[i]==(1601+offset3))
{
	vR[i] = 1601;
}
else
{
	vR[i] = 1600;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1567+offset3))
{
	if(input[i]>(1583+offset3))
{
	if(input[i]>(1591+offset3))
{
	if(input[i]>(1595+offset3))
{
	if(input[i]>(1597+offset3))
{
	if(input[i]==(1599+offset3))
{
	vR[i] = 1599;
}
else
{
	vR[i] = 1598;
}

}
else
{
	if(input[i]==(1597+offset3))
{
	vR[i] = 1597;
}
else
{
	vR[i] = 1596;
}

}

}
else
{
	if(input[i]>(1593+offset3))
{
	if(input[i]==(1595+offset3))
{
	vR[i] = 1595;
}
else
{
	vR[i] = 1594;
}

}
else
{
	if(input[i]==(1593+offset3))
{
	vR[i] = 1593;
}
else
{
	vR[i] = 1592;
}

}

}

}
else
{
	if(input[i]>(1587+offset3))
{
	if(input[i]>(1589+offset3))
{
	if(input[i]==(1591+offset3))
{
	vR[i] = 1591;
}
else
{
	vR[i] = 1590;
}

}
else
{
	if(input[i]==(1589+offset3))
{
	vR[i] = 1589;
}
else
{
	vR[i] = 1588;
}

}

}
else
{
	if(input[i]>(1585+offset3))
{
	if(input[i]==(1587+offset3))
{
	vR[i] = 1587;
}
else
{
	vR[i] = 1586;
}

}
else
{
	if(input[i]==(1585+offset3))
{
	vR[i] = 1585;
}
else
{
	vR[i] = 1584;
}

}

}

}

}
else
{
	if(input[i]>(1575+offset3))
{
	if(input[i]>(1579+offset3))
{
	if(input[i]>(1581+offset3))
{
	if(input[i]==(1583+offset3))
{
	vR[i] = 1583;
}
else
{
	vR[i] = 1582;
}

}
else
{
	if(input[i]==(1581+offset3))
{
	vR[i] = 1581;
}
else
{
	vR[i] = 1580;
}

}

}
else
{
	if(input[i]>(1577+offset3))
{
	if(input[i]==(1579+offset3))
{
	vR[i] = 1579;
}
else
{
	vR[i] = 1578;
}

}
else
{
	if(input[i]==(1577+offset3))
{
	vR[i] = 1577;
}
else
{
	vR[i] = 1576;
}

}

}

}
else
{
	if(input[i]>(1571+offset3))
{
	if(input[i]>(1573+offset3))
{
	if(input[i]==(1575+offset3))
{
	vR[i] = 1575;
}
else
{
	vR[i] = 1574;
}

}
else
{
	if(input[i]==(1573+offset3))
{
	vR[i] = 1573;
}
else
{
	vR[i] = 1572;
}

}

}
else
{
	if(input[i]>(1569+offset3))
{
	if(input[i]==(1571+offset3))
{
	vR[i] = 1571;
}
else
{
	vR[i] = 1570;
}

}
else
{
	if(input[i]==(1569+offset3))
{
	vR[i] = 1569;
}
else
{
	vR[i] = 1568;
}

}

}

}

}

}
else
{
	if(input[i]>(1551+offset3))
{
	if(input[i]>(1559+offset3))
{
	if(input[i]>(1563+offset3))
{
	if(input[i]>(1565+offset3))
{
	if(input[i]==(1567+offset3))
{
	vR[i] = 1567;
}
else
{
	vR[i] = 1566;
}

}
else
{
	if(input[i]==(1565+offset3))
{
	vR[i] = 1565;
}
else
{
	vR[i] = 1564;
}

}

}
else
{
	if(input[i]>(1561+offset3))
{
	if(input[i]==(1563+offset3))
{
	vR[i] = 1563;
}
else
{
	vR[i] = 1562;
}

}
else
{
	if(input[i]==(1561+offset3))
{
	vR[i] = 1561;
}
else
{
	vR[i] = 1560;
}

}

}

}
else
{
	if(input[i]>(1555+offset3))
{
	if(input[i]>(1557+offset3))
{
	if(input[i]==(1559+offset3))
{
	vR[i] = 1559;
}
else
{
	vR[i] = 1558;
}

}
else
{
	if(input[i]==(1557+offset3))
{
	vR[i] = 1557;
}
else
{
	vR[i] = 1556;
}

}

}
else
{
	if(input[i]>(1553+offset3))
{
	if(input[i]==(1555+offset3))
{
	vR[i] = 1555;
}
else
{
	vR[i] = 1554;
}

}
else
{
	if(input[i]==(1553+offset3))
{
	vR[i] = 1553;
}
else
{
	vR[i] = 1552;
}

}

}

}

}
else
{
	if(input[i]>(1543+offset3))
{
	if(input[i]>(1547+offset3))
{
	if(input[i]>(1549+offset3))
{
	if(input[i]==(1551+offset3))
{
	vR[i] = 1551;
}
else
{
	vR[i] = 1550;
}

}
else
{
	if(input[i]==(1549+offset3))
{
	vR[i] = 1549;
}
else
{
	vR[i] = 1548;
}

}

}
else
{
	if(input[i]>(1545+offset3))
{
	if(input[i]==(1547+offset3))
{
	vR[i] = 1547;
}
else
{
	vR[i] = 1546;
}

}
else
{
	if(input[i]==(1545+offset3))
{
	vR[i] = 1545;
}
else
{
	vR[i] = 1544;
}

}

}

}
else
{
	if(input[i]>(1539+offset3))
{
	if(input[i]>(1541+offset3))
{
	if(input[i]==(1543+offset3))
{
	vR[i] = 1543;
}
else
{
	vR[i] = 1542;
}

}
else
{
	if(input[i]==(1541+offset3))
{
	vR[i] = 1541;
}
else
{
	vR[i] = 1540;
}

}

}
else
{
	if(input[i]>(1537+offset3))
{
	if(input[i]==(1539+offset3))
{
	vR[i] = 1539;
}
else
{
	vR[i] = 1538;
}

}
else
{
	if(input[i]==(1537+offset3))
{
	vR[i] = 1537;
}
else
{
	vR[i] = 1536;
}

}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(1279+offset3))
{
	if(input[i]>(1407+offset3))
{
	if(input[i]>(1471+offset3))
{
	if(input[i]>(1503+offset3))
{
	if(input[i]>(1519+offset3))
{
	if(input[i]>(1527+offset3))
{
	if(input[i]>(1531+offset3))
{
	if(input[i]>(1533+offset3))
{
	if(input[i]==(1535+offset3))
{
	vR[i] = 1535;
}
else
{
	vR[i] = 1534;
}

}
else
{
	if(input[i]==(1533+offset3))
{
	vR[i] = 1533;
}
else
{
	vR[i] = 1532;
}

}

}
else
{
	if(input[i]>(1529+offset3))
{
	if(input[i]==(1531+offset3))
{
	vR[i] = 1531;
}
else
{
	vR[i] = 1530;
}

}
else
{
	if(input[i]==(1529+offset3))
{
	vR[i] = 1529;
}
else
{
	vR[i] = 1528;
}

}

}

}
else
{
	if(input[i]>(1523+offset3))
{
	if(input[i]>(1525+offset3))
{
	if(input[i]==(1527+offset3))
{
	vR[i] = 1527;
}
else
{
	vR[i] = 1526;
}

}
else
{
	if(input[i]==(1525+offset3))
{
	vR[i] = 1525;
}
else
{
	vR[i] = 1524;
}

}

}
else
{
	if(input[i]>(1521+offset3))
{
	if(input[i]==(1523+offset3))
{
	vR[i] = 1523;
}
else
{
	vR[i] = 1522;
}

}
else
{
	if(input[i]==(1521+offset3))
{
	vR[i] = 1521;
}
else
{
	vR[i] = 1520;
}

}

}

}

}
else
{
	if(input[i]>(1511+offset3))
{
	if(input[i]>(1515+offset3))
{
	if(input[i]>(1517+offset3))
{
	if(input[i]==(1519+offset3))
{
	vR[i] = 1519;
}
else
{
	vR[i] = 1518;
}

}
else
{
	if(input[i]==(1517+offset3))
{
	vR[i] = 1517;
}
else
{
	vR[i] = 1516;
}

}

}
else
{
	if(input[i]>(1513+offset3))
{
	if(input[i]==(1515+offset3))
{
	vR[i] = 1515;
}
else
{
	vR[i] = 1514;
}

}
else
{
	if(input[i]==(1513+offset3))
{
	vR[i] = 1513;
}
else
{
	vR[i] = 1512;
}

}

}

}
else
{
	if(input[i]>(1507+offset3))
{
	if(input[i]>(1509+offset3))
{
	if(input[i]==(1511+offset3))
{
	vR[i] = 1511;
}
else
{
	vR[i] = 1510;
}

}
else
{
	if(input[i]==(1509+offset3))
{
	vR[i] = 1509;
}
else
{
	vR[i] = 1508;
}

}

}
else
{
	if(input[i]>(1505+offset3))
{
	if(input[i]==(1507+offset3))
{
	vR[i] = 1507;
}
else
{
	vR[i] = 1506;
}

}
else
{
	if(input[i]==(1505+offset3))
{
	vR[i] = 1505;
}
else
{
	vR[i] = 1504;
}

}

}

}

}

}
else
{
	if(input[i]>(1487+offset3))
{
	if(input[i]>(1495+offset3))
{
	if(input[i]>(1499+offset3))
{
	if(input[i]>(1501+offset3))
{
	if(input[i]==(1503+offset3))
{
	vR[i] = 1503;
}
else
{
	vR[i] = 1502;
}

}
else
{
	if(input[i]==(1501+offset3))
{
	vR[i] = 1501;
}
else
{
	vR[i] = 1500;
}

}

}
else
{
	if(input[i]>(1497+offset3))
{
	if(input[i]==(1499+offset3))
{
	vR[i] = 1499;
}
else
{
	vR[i] = 1498;
}

}
else
{
	if(input[i]==(1497+offset3))
{
	vR[i] = 1497;
}
else
{
	vR[i] = 1496;
}

}

}

}
else
{
	if(input[i]>(1491+offset3))
{
	if(input[i]>(1493+offset3))
{
	if(input[i]==(1495+offset3))
{
	vR[i] = 1495;
}
else
{
	vR[i] = 1494;
}

}
else
{
	if(input[i]==(1493+offset3))
{
	vR[i] = 1493;
}
else
{
	vR[i] = 1492;
}

}

}
else
{
	if(input[i]>(1489+offset3))
{
	if(input[i]==(1491+offset3))
{
	vR[i] = 1491;
}
else
{
	vR[i] = 1490;
}

}
else
{
	if(input[i]==(1489+offset3))
{
	vR[i] = 1489;
}
else
{
	vR[i] = 1488;
}

}

}

}

}
else
{
	if(input[i]>(1479+offset3))
{
	if(input[i]>(1483+offset3))
{
	if(input[i]>(1485+offset3))
{
	if(input[i]==(1487+offset3))
{
	vR[i] = 1487;
}
else
{
	vR[i] = 1486;
}

}
else
{
	if(input[i]==(1485+offset3))
{
	vR[i] = 1485;
}
else
{
	vR[i] = 1484;
}

}

}
else
{
	if(input[i]>(1481+offset3))
{
	if(input[i]==(1483+offset3))
{
	vR[i] = 1483;
}
else
{
	vR[i] = 1482;
}

}
else
{
	if(input[i]==(1481+offset3))
{
	vR[i] = 1481;
}
else
{
	vR[i] = 1480;
}

}

}

}
else
{
	if(input[i]>(1475+offset3))
{
	if(input[i]>(1477+offset3))
{
	if(input[i]==(1479+offset3))
{
	vR[i] = 1479;
}
else
{
	vR[i] = 1478;
}

}
else
{
	if(input[i]==(1477+offset3))
{
	vR[i] = 1477;
}
else
{
	vR[i] = 1476;
}

}

}
else
{
	if(input[i]>(1473+offset3))
{
	if(input[i]==(1475+offset3))
{
	vR[i] = 1475;
}
else
{
	vR[i] = 1474;
}

}
else
{
	if(input[i]==(1473+offset3))
{
	vR[i] = 1473;
}
else
{
	vR[i] = 1472;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1439+offset3))
{
	if(input[i]>(1455+offset3))
{
	if(input[i]>(1463+offset3))
{
	if(input[i]>(1467+offset3))
{
	if(input[i]>(1469+offset3))
{
	if(input[i]==(1471+offset3))
{
	vR[i] = 1471;
}
else
{
	vR[i] = 1470;
}

}
else
{
	if(input[i]==(1469+offset3))
{
	vR[i] = 1469;
}
else
{
	vR[i] = 1468;
}

}

}
else
{
	if(input[i]>(1465+offset3))
{
	if(input[i]==(1467+offset3))
{
	vR[i] = 1467;
}
else
{
	vR[i] = 1466;
}

}
else
{
	if(input[i]==(1465+offset3))
{
	vR[i] = 1465;
}
else
{
	vR[i] = 1464;
}

}

}

}
else
{
	if(input[i]>(1459+offset3))
{
	if(input[i]>(1461+offset3))
{
	if(input[i]==(1463+offset3))
{
	vR[i] = 1463;
}
else
{
	vR[i] = 1462;
}

}
else
{
	if(input[i]==(1461+offset3))
{
	vR[i] = 1461;
}
else
{
	vR[i] = 1460;
}

}

}
else
{
	if(input[i]>(1457+offset3))
{
	if(input[i]==(1459+offset3))
{
	vR[i] = 1459;
}
else
{
	vR[i] = 1458;
}

}
else
{
	if(input[i]==(1457+offset3))
{
	vR[i] = 1457;
}
else
{
	vR[i] = 1456;
}

}

}

}

}
else
{
	if(input[i]>(1447+offset3))
{
	if(input[i]>(1451+offset3))
{
	if(input[i]>(1453+offset3))
{
	if(input[i]==(1455+offset3))
{
	vR[i] = 1455;
}
else
{
	vR[i] = 1454;
}

}
else
{
	if(input[i]==(1453+offset3))
{
	vR[i] = 1453;
}
else
{
	vR[i] = 1452;
}

}

}
else
{
	if(input[i]>(1449+offset3))
{
	if(input[i]==(1451+offset3))
{
	vR[i] = 1451;
}
else
{
	vR[i] = 1450;
}

}
else
{
	if(input[i]==(1449+offset3))
{
	vR[i] = 1449;
}
else
{
	vR[i] = 1448;
}

}

}

}
else
{
	if(input[i]>(1443+offset3))
{
	if(input[i]>(1445+offset3))
{
	if(input[i]==(1447+offset3))
{
	vR[i] = 1447;
}
else
{
	vR[i] = 1446;
}

}
else
{
	if(input[i]==(1445+offset3))
{
	vR[i] = 1445;
}
else
{
	vR[i] = 1444;
}

}

}
else
{
	if(input[i]>(1441+offset3))
{
	if(input[i]==(1443+offset3))
{
	vR[i] = 1443;
}
else
{
	vR[i] = 1442;
}

}
else
{
	if(input[i]==(1441+offset3))
{
	vR[i] = 1441;
}
else
{
	vR[i] = 1440;
}

}

}

}

}

}
else
{
	if(input[i]>(1423+offset3))
{
	if(input[i]>(1431+offset3))
{
	if(input[i]>(1435+offset3))
{
	if(input[i]>(1437+offset3))
{
	if(input[i]==(1439+offset3))
{
	vR[i] = 1439;
}
else
{
	vR[i] = 1438;
}

}
else
{
	if(input[i]==(1437+offset3))
{
	vR[i] = 1437;
}
else
{
	vR[i] = 1436;
}

}

}
else
{
	if(input[i]>(1433+offset3))
{
	if(input[i]==(1435+offset3))
{
	vR[i] = 1435;
}
else
{
	vR[i] = 1434;
}

}
else
{
	if(input[i]==(1433+offset3))
{
	vR[i] = 1433;
}
else
{
	vR[i] = 1432;
}

}

}

}
else
{
	if(input[i]>(1427+offset3))
{
	if(input[i]>(1429+offset3))
{
	if(input[i]==(1431+offset3))
{
	vR[i] = 1431;
}
else
{
	vR[i] = 1430;
}

}
else
{
	if(input[i]==(1429+offset3))
{
	vR[i] = 1429;
}
else
{
	vR[i] = 1428;
}

}

}
else
{
	if(input[i]>(1425+offset3))
{
	if(input[i]==(1427+offset3))
{
	vR[i] = 1427;
}
else
{
	vR[i] = 1426;
}

}
else
{
	if(input[i]==(1425+offset3))
{
	vR[i] = 1425;
}
else
{
	vR[i] = 1424;
}

}

}

}

}
else
{
	if(input[i]>(1415+offset3))
{
	if(input[i]>(1419+offset3))
{
	if(input[i]>(1421+offset3))
{
	if(input[i]==(1423+offset3))
{
	vR[i] = 1423;
}
else
{
	vR[i] = 1422;
}

}
else
{
	if(input[i]==(1421+offset3))
{
	vR[i] = 1421;
}
else
{
	vR[i] = 1420;
}

}

}
else
{
	if(input[i]>(1417+offset3))
{
	if(input[i]==(1419+offset3))
{
	vR[i] = 1419;
}
else
{
	vR[i] = 1418;
}

}
else
{
	if(input[i]==(1417+offset3))
{
	vR[i] = 1417;
}
else
{
	vR[i] = 1416;
}

}

}

}
else
{
	if(input[i]>(1411+offset3))
{
	if(input[i]>(1413+offset3))
{
	if(input[i]==(1415+offset3))
{
	vR[i] = 1415;
}
else
{
	vR[i] = 1414;
}

}
else
{
	if(input[i]==(1413+offset3))
{
	vR[i] = 1413;
}
else
{
	vR[i] = 1412;
}

}

}
else
{
	if(input[i]>(1409+offset3))
{
	if(input[i]==(1411+offset3))
{
	vR[i] = 1411;
}
else
{
	vR[i] = 1410;
}

}
else
{
	if(input[i]==(1409+offset3))
{
	vR[i] = 1409;
}
else
{
	vR[i] = 1408;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(1343+offset3))
{
	if(input[i]>(1375+offset3))
{
	if(input[i]>(1391+offset3))
{
	if(input[i]>(1399+offset3))
{
	if(input[i]>(1403+offset3))
{
	if(input[i]>(1405+offset3))
{
	if(input[i]==(1407+offset3))
{
	vR[i] = 1407;
}
else
{
	vR[i] = 1406;
}

}
else
{
	if(input[i]==(1405+offset3))
{
	vR[i] = 1405;
}
else
{
	vR[i] = 1404;
}

}

}
else
{
	if(input[i]>(1401+offset3))
{
	if(input[i]==(1403+offset3))
{
	vR[i] = 1403;
}
else
{
	vR[i] = 1402;
}

}
else
{
	if(input[i]==(1401+offset3))
{
	vR[i] = 1401;
}
else
{
	vR[i] = 1400;
}

}

}

}
else
{
	if(input[i]>(1395+offset3))
{
	if(input[i]>(1397+offset3))
{
	if(input[i]==(1399+offset3))
{
	vR[i] = 1399;
}
else
{
	vR[i] = 1398;
}

}
else
{
	if(input[i]==(1397+offset3))
{
	vR[i] = 1397;
}
else
{
	vR[i] = 1396;
}

}

}
else
{
	if(input[i]>(1393+offset3))
{
	if(input[i]==(1395+offset3))
{
	vR[i] = 1395;
}
else
{
	vR[i] = 1394;
}

}
else
{
	if(input[i]==(1393+offset3))
{
	vR[i] = 1393;
}
else
{
	vR[i] = 1392;
}

}

}

}

}
else
{
	if(input[i]>(1383+offset3))
{
	if(input[i]>(1387+offset3))
{
	if(input[i]>(1389+offset3))
{
	if(input[i]==(1391+offset3))
{
	vR[i] = 1391;
}
else
{
	vR[i] = 1390;
}

}
else
{
	if(input[i]==(1389+offset3))
{
	vR[i] = 1389;
}
else
{
	vR[i] = 1388;
}

}

}
else
{
	if(input[i]>(1385+offset3))
{
	if(input[i]==(1387+offset3))
{
	vR[i] = 1387;
}
else
{
	vR[i] = 1386;
}

}
else
{
	if(input[i]==(1385+offset3))
{
	vR[i] = 1385;
}
else
{
	vR[i] = 1384;
}

}

}

}
else
{
	if(input[i]>(1379+offset3))
{
	if(input[i]>(1381+offset3))
{
	if(input[i]==(1383+offset3))
{
	vR[i] = 1383;
}
else
{
	vR[i] = 1382;
}

}
else
{
	if(input[i]==(1381+offset3))
{
	vR[i] = 1381;
}
else
{
	vR[i] = 1380;
}

}

}
else
{
	if(input[i]>(1377+offset3))
{
	if(input[i]==(1379+offset3))
{
	vR[i] = 1379;
}
else
{
	vR[i] = 1378;
}

}
else
{
	if(input[i]==(1377+offset3))
{
	vR[i] = 1377;
}
else
{
	vR[i] = 1376;
}

}

}

}

}

}
else
{
	if(input[i]>(1359+offset3))
{
	if(input[i]>(1367+offset3))
{
	if(input[i]>(1371+offset3))
{
	if(input[i]>(1373+offset3))
{
	if(input[i]==(1375+offset3))
{
	vR[i] = 1375;
}
else
{
	vR[i] = 1374;
}

}
else
{
	if(input[i]==(1373+offset3))
{
	vR[i] = 1373;
}
else
{
	vR[i] = 1372;
}

}

}
else
{
	if(input[i]>(1369+offset3))
{
	if(input[i]==(1371+offset3))
{
	vR[i] = 1371;
}
else
{
	vR[i] = 1370;
}

}
else
{
	if(input[i]==(1369+offset3))
{
	vR[i] = 1369;
}
else
{
	vR[i] = 1368;
}

}

}

}
else
{
	if(input[i]>(1363+offset3))
{
	if(input[i]>(1365+offset3))
{
	if(input[i]==(1367+offset3))
{
	vR[i] = 1367;
}
else
{
	vR[i] = 1366;
}

}
else
{
	if(input[i]==(1365+offset3))
{
	vR[i] = 1365;
}
else
{
	vR[i] = 1364;
}

}

}
else
{
	if(input[i]>(1361+offset3))
{
	if(input[i]==(1363+offset3))
{
	vR[i] = 1363;
}
else
{
	vR[i] = 1362;
}

}
else
{
	if(input[i]==(1361+offset3))
{
	vR[i] = 1361;
}
else
{
	vR[i] = 1360;
}

}

}

}

}
else
{
	if(input[i]>(1351+offset3))
{
	if(input[i]>(1355+offset3))
{
	if(input[i]>(1357+offset3))
{
	if(input[i]==(1359+offset3))
{
	vR[i] = 1359;
}
else
{
	vR[i] = 1358;
}

}
else
{
	if(input[i]==(1357+offset3))
{
	vR[i] = 1357;
}
else
{
	vR[i] = 1356;
}

}

}
else
{
	if(input[i]>(1353+offset3))
{
	if(input[i]==(1355+offset3))
{
	vR[i] = 1355;
}
else
{
	vR[i] = 1354;
}

}
else
{
	if(input[i]==(1353+offset3))
{
	vR[i] = 1353;
}
else
{
	vR[i] = 1352;
}

}

}

}
else
{
	if(input[i]>(1347+offset3))
{
	if(input[i]>(1349+offset3))
{
	if(input[i]==(1351+offset3))
{
	vR[i] = 1351;
}
else
{
	vR[i] = 1350;
}

}
else
{
	if(input[i]==(1349+offset3))
{
	vR[i] = 1349;
}
else
{
	vR[i] = 1348;
}

}

}
else
{
	if(input[i]>(1345+offset3))
{
	if(input[i]==(1347+offset3))
{
	vR[i] = 1347;
}
else
{
	vR[i] = 1346;
}

}
else
{
	if(input[i]==(1345+offset3))
{
	vR[i] = 1345;
}
else
{
	vR[i] = 1344;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1311+offset3))
{
	if(input[i]>(1327+offset3))
{
	if(input[i]>(1335+offset3))
{
	if(input[i]>(1339+offset3))
{
	if(input[i]>(1341+offset3))
{
	if(input[i]==(1343+offset3))
{
	vR[i] = 1343;
}
else
{
	vR[i] = 1342;
}

}
else
{
	if(input[i]==(1341+offset3))
{
	vR[i] = 1341;
}
else
{
	vR[i] = 1340;
}

}

}
else
{
	if(input[i]>(1337+offset3))
{
	if(input[i]==(1339+offset3))
{
	vR[i] = 1339;
}
else
{
	vR[i] = 1338;
}

}
else
{
	if(input[i]==(1337+offset3))
{
	vR[i] = 1337;
}
else
{
	vR[i] = 1336;
}

}

}

}
else
{
	if(input[i]>(1331+offset3))
{
	if(input[i]>(1333+offset3))
{
	if(input[i]==(1335+offset3))
{
	vR[i] = 1335;
}
else
{
	vR[i] = 1334;
}

}
else
{
	if(input[i]==(1333+offset3))
{
	vR[i] = 1333;
}
else
{
	vR[i] = 1332;
}

}

}
else
{
	if(input[i]>(1329+offset3))
{
	if(input[i]==(1331+offset3))
{
	vR[i] = 1331;
}
else
{
	vR[i] = 1330;
}

}
else
{
	if(input[i]==(1329+offset3))
{
	vR[i] = 1329;
}
else
{
	vR[i] = 1328;
}

}

}

}

}
else
{
	if(input[i]>(1319+offset3))
{
	if(input[i]>(1323+offset3))
{
	if(input[i]>(1325+offset3))
{
	if(input[i]==(1327+offset3))
{
	vR[i] = 1327;
}
else
{
	vR[i] = 1326;
}

}
else
{
	if(input[i]==(1325+offset3))
{
	vR[i] = 1325;
}
else
{
	vR[i] = 1324;
}

}

}
else
{
	if(input[i]>(1321+offset3))
{
	if(input[i]==(1323+offset3))
{
	vR[i] = 1323;
}
else
{
	vR[i] = 1322;
}

}
else
{
	if(input[i]==(1321+offset3))
{
	vR[i] = 1321;
}
else
{
	vR[i] = 1320;
}

}

}

}
else
{
	if(input[i]>(1315+offset3))
{
	if(input[i]>(1317+offset3))
{
	if(input[i]==(1319+offset3))
{
	vR[i] = 1319;
}
else
{
	vR[i] = 1318;
}

}
else
{
	if(input[i]==(1317+offset3))
{
	vR[i] = 1317;
}
else
{
	vR[i] = 1316;
}

}

}
else
{
	if(input[i]>(1313+offset3))
{
	if(input[i]==(1315+offset3))
{
	vR[i] = 1315;
}
else
{
	vR[i] = 1314;
}

}
else
{
	if(input[i]==(1313+offset3))
{
	vR[i] = 1313;
}
else
{
	vR[i] = 1312;
}

}

}

}

}

}
else
{
	if(input[i]>(1295+offset3))
{
	if(input[i]>(1303+offset3))
{
	if(input[i]>(1307+offset3))
{
	if(input[i]>(1309+offset3))
{
	if(input[i]==(1311+offset3))
{
	vR[i] = 1311;
}
else
{
	vR[i] = 1310;
}

}
else
{
	if(input[i]==(1309+offset3))
{
	vR[i] = 1309;
}
else
{
	vR[i] = 1308;
}

}

}
else
{
	if(input[i]>(1305+offset3))
{
	if(input[i]==(1307+offset3))
{
	vR[i] = 1307;
}
else
{
	vR[i] = 1306;
}

}
else
{
	if(input[i]==(1305+offset3))
{
	vR[i] = 1305;
}
else
{
	vR[i] = 1304;
}

}

}

}
else
{
	if(input[i]>(1299+offset3))
{
	if(input[i]>(1301+offset3))
{
	if(input[i]==(1303+offset3))
{
	vR[i] = 1303;
}
else
{
	vR[i] = 1302;
}

}
else
{
	if(input[i]==(1301+offset3))
{
	vR[i] = 1301;
}
else
{
	vR[i] = 1300;
}

}

}
else
{
	if(input[i]>(1297+offset3))
{
	if(input[i]==(1299+offset3))
{
	vR[i] = 1299;
}
else
{
	vR[i] = 1298;
}

}
else
{
	if(input[i]==(1297+offset3))
{
	vR[i] = 1297;
}
else
{
	vR[i] = 1296;
}

}

}

}

}
else
{
	if(input[i]>(1287+offset3))
{
	if(input[i]>(1291+offset3))
{
	if(input[i]>(1293+offset3))
{
	if(input[i]==(1295+offset3))
{
	vR[i] = 1295;
}
else
{
	vR[i] = 1294;
}

}
else
{
	if(input[i]==(1293+offset3))
{
	vR[i] = 1293;
}
else
{
	vR[i] = 1292;
}

}

}
else
{
	if(input[i]>(1289+offset3))
{
	if(input[i]==(1291+offset3))
{
	vR[i] = 1291;
}
else
{
	vR[i] = 1290;
}

}
else
{
	if(input[i]==(1289+offset3))
{
	vR[i] = 1289;
}
else
{
	vR[i] = 1288;
}

}

}

}
else
{
	if(input[i]>(1283+offset3))
{
	if(input[i]>(1285+offset3))
{
	if(input[i]==(1287+offset3))
{
	vR[i] = 1287;
}
else
{
	vR[i] = 1286;
}

}
else
{
	if(input[i]==(1285+offset3))
{
	vR[i] = 1285;
}
else
{
	vR[i] = 1284;
}

}

}
else
{
	if(input[i]>(1281+offset3))
{
	if(input[i]==(1283+offset3))
{
	vR[i] = 1283;
}
else
{
	vR[i] = 1282;
}

}
else
{
	if(input[i]==(1281+offset3))
{
	vR[i] = 1281;
}
else
{
	vR[i] = 1280;
}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(1151+offset3))
{
	if(input[i]>(1215+offset3))
{
	if(input[i]>(1247+offset3))
{
	if(input[i]>(1263+offset3))
{
	if(input[i]>(1271+offset3))
{
	if(input[i]>(1275+offset3))
{
	if(input[i]>(1277+offset3))
{
	if(input[i]==(1279+offset3))
{
	vR[i] = 1279;
}
else
{
	vR[i] = 1278;
}

}
else
{
	if(input[i]==(1277+offset3))
{
	vR[i] = 1277;
}
else
{
	vR[i] = 1276;
}

}

}
else
{
	if(input[i]>(1273+offset3))
{
	if(input[i]==(1275+offset3))
{
	vR[i] = 1275;
}
else
{
	vR[i] = 1274;
}

}
else
{
	if(input[i]==(1273+offset3))
{
	vR[i] = 1273;
}
else
{
	vR[i] = 1272;
}

}

}

}
else
{
	if(input[i]>(1267+offset3))
{
	if(input[i]>(1269+offset3))
{
	if(input[i]==(1271+offset3))
{
	vR[i] = 1271;
}
else
{
	vR[i] = 1270;
}

}
else
{
	if(input[i]==(1269+offset3))
{
	vR[i] = 1269;
}
else
{
	vR[i] = 1268;
}

}

}
else
{
	if(input[i]>(1265+offset3))
{
	if(input[i]==(1267+offset3))
{
	vR[i] = 1267;
}
else
{
	vR[i] = 1266;
}

}
else
{
	if(input[i]==(1265+offset3))
{
	vR[i] = 1265;
}
else
{
	vR[i] = 1264;
}

}

}

}

}
else
{
	if(input[i]>(1255+offset3))
{
	if(input[i]>(1259+offset3))
{
	if(input[i]>(1261+offset3))
{
	if(input[i]==(1263+offset3))
{
	vR[i] = 1263;
}
else
{
	vR[i] = 1262;
}

}
else
{
	if(input[i]==(1261+offset3))
{
	vR[i] = 1261;
}
else
{
	vR[i] = 1260;
}

}

}
else
{
	if(input[i]>(1257+offset3))
{
	if(input[i]==(1259+offset3))
{
	vR[i] = 1259;
}
else
{
	vR[i] = 1258;
}

}
else
{
	if(input[i]==(1257+offset3))
{
	vR[i] = 1257;
}
else
{
	vR[i] = 1256;
}

}

}

}
else
{
	if(input[i]>(1251+offset3))
{
	if(input[i]>(1253+offset3))
{
	if(input[i]==(1255+offset3))
{
	vR[i] = 1255;
}
else
{
	vR[i] = 1254;
}

}
else
{
	if(input[i]==(1253+offset3))
{
	vR[i] = 1253;
}
else
{
	vR[i] = 1252;
}

}

}
else
{
	if(input[i]>(1249+offset3))
{
	if(input[i]==(1251+offset3))
{
	vR[i] = 1251;
}
else
{
	vR[i] = 1250;
}

}
else
{
	if(input[i]==(1249+offset3))
{
	vR[i] = 1249;
}
else
{
	vR[i] = 1248;
}

}

}

}

}

}
else
{
	if(input[i]>(1231+offset3))
{
	if(input[i]>(1239+offset3))
{
	if(input[i]>(1243+offset3))
{
	if(input[i]>(1245+offset3))
{
	if(input[i]==(1247+offset3))
{
	vR[i] = 1247;
}
else
{
	vR[i] = 1246;
}

}
else
{
	if(input[i]==(1245+offset3))
{
	vR[i] = 1245;
}
else
{
	vR[i] = 1244;
}

}

}
else
{
	if(input[i]>(1241+offset3))
{
	if(input[i]==(1243+offset3))
{
	vR[i] = 1243;
}
else
{
	vR[i] = 1242;
}

}
else
{
	if(input[i]==(1241+offset3))
{
	vR[i] = 1241;
}
else
{
	vR[i] = 1240;
}

}

}

}
else
{
	if(input[i]>(1235+offset3))
{
	if(input[i]>(1237+offset3))
{
	if(input[i]==(1239+offset3))
{
	vR[i] = 1239;
}
else
{
	vR[i] = 1238;
}

}
else
{
	if(input[i]==(1237+offset3))
{
	vR[i] = 1237;
}
else
{
	vR[i] = 1236;
}

}

}
else
{
	if(input[i]>(1233+offset3))
{
	if(input[i]==(1235+offset3))
{
	vR[i] = 1235;
}
else
{
	vR[i] = 1234;
}

}
else
{
	if(input[i]==(1233+offset3))
{
	vR[i] = 1233;
}
else
{
	vR[i] = 1232;
}

}

}

}

}
else
{
	if(input[i]>(1223+offset3))
{
	if(input[i]>(1227+offset3))
{
	if(input[i]>(1229+offset3))
{
	if(input[i]==(1231+offset3))
{
	vR[i] = 1231;
}
else
{
	vR[i] = 1230;
}

}
else
{
	if(input[i]==(1229+offset3))
{
	vR[i] = 1229;
}
else
{
	vR[i] = 1228;
}

}

}
else
{
	if(input[i]>(1225+offset3))
{
	if(input[i]==(1227+offset3))
{
	vR[i] = 1227;
}
else
{
	vR[i] = 1226;
}

}
else
{
	if(input[i]==(1225+offset3))
{
	vR[i] = 1225;
}
else
{
	vR[i] = 1224;
}

}

}

}
else
{
	if(input[i]>(1219+offset3))
{
	if(input[i]>(1221+offset3))
{
	if(input[i]==(1223+offset3))
{
	vR[i] = 1223;
}
else
{
	vR[i] = 1222;
}

}
else
{
	if(input[i]==(1221+offset3))
{
	vR[i] = 1221;
}
else
{
	vR[i] = 1220;
}

}

}
else
{
	if(input[i]>(1217+offset3))
{
	if(input[i]==(1219+offset3))
{
	vR[i] = 1219;
}
else
{
	vR[i] = 1218;
}

}
else
{
	if(input[i]==(1217+offset3))
{
	vR[i] = 1217;
}
else
{
	vR[i] = 1216;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1183+offset3))
{
	if(input[i]>(1199+offset3))
{
	if(input[i]>(1207+offset3))
{
	if(input[i]>(1211+offset3))
{
	if(input[i]>(1213+offset3))
{
	if(input[i]==(1215+offset3))
{
	vR[i] = 1215;
}
else
{
	vR[i] = 1214;
}

}
else
{
	if(input[i]==(1213+offset3))
{
	vR[i] = 1213;
}
else
{
	vR[i] = 1212;
}

}

}
else
{
	if(input[i]>(1209+offset3))
{
	if(input[i]==(1211+offset3))
{
	vR[i] = 1211;
}
else
{
	vR[i] = 1210;
}

}
else
{
	if(input[i]==(1209+offset3))
{
	vR[i] = 1209;
}
else
{
	vR[i] = 1208;
}

}

}

}
else
{
	if(input[i]>(1203+offset3))
{
	if(input[i]>(1205+offset3))
{
	if(input[i]==(1207+offset3))
{
	vR[i] = 1207;
}
else
{
	vR[i] = 1206;
}

}
else
{
	if(input[i]==(1205+offset3))
{
	vR[i] = 1205;
}
else
{
	vR[i] = 1204;
}

}

}
else
{
	if(input[i]>(1201+offset3))
{
	if(input[i]==(1203+offset3))
{
	vR[i] = 1203;
}
else
{
	vR[i] = 1202;
}

}
else
{
	if(input[i]==(1201+offset3))
{
	vR[i] = 1201;
}
else
{
	vR[i] = 1200;
}

}

}

}

}
else
{
	if(input[i]>(1191+offset3))
{
	if(input[i]>(1195+offset3))
{
	if(input[i]>(1197+offset3))
{
	if(input[i]==(1199+offset3))
{
	vR[i] = 1199;
}
else
{
	vR[i] = 1198;
}

}
else
{
	if(input[i]==(1197+offset3))
{
	vR[i] = 1197;
}
else
{
	vR[i] = 1196;
}

}

}
else
{
	if(input[i]>(1193+offset3))
{
	if(input[i]==(1195+offset3))
{
	vR[i] = 1195;
}
else
{
	vR[i] = 1194;
}

}
else
{
	if(input[i]==(1193+offset3))
{
	vR[i] = 1193;
}
else
{
	vR[i] = 1192;
}

}

}

}
else
{
	if(input[i]>(1187+offset3))
{
	if(input[i]>(1189+offset3))
{
	if(input[i]==(1191+offset3))
{
	vR[i] = 1191;
}
else
{
	vR[i] = 1190;
}

}
else
{
	if(input[i]==(1189+offset3))
{
	vR[i] = 1189;
}
else
{
	vR[i] = 1188;
}

}

}
else
{
	if(input[i]>(1185+offset3))
{
	if(input[i]==(1187+offset3))
{
	vR[i] = 1187;
}
else
{
	vR[i] = 1186;
}

}
else
{
	if(input[i]==(1185+offset3))
{
	vR[i] = 1185;
}
else
{
	vR[i] = 1184;
}

}

}

}

}

}
else
{
	if(input[i]>(1167+offset3))
{
	if(input[i]>(1175+offset3))
{
	if(input[i]>(1179+offset3))
{
	if(input[i]>(1181+offset3))
{
	if(input[i]==(1183+offset3))
{
	vR[i] = 1183;
}
else
{
	vR[i] = 1182;
}

}
else
{
	if(input[i]==(1181+offset3))
{
	vR[i] = 1181;
}
else
{
	vR[i] = 1180;
}

}

}
else
{
	if(input[i]>(1177+offset3))
{
	if(input[i]==(1179+offset3))
{
	vR[i] = 1179;
}
else
{
	vR[i] = 1178;
}

}
else
{
	if(input[i]==(1177+offset3))
{
	vR[i] = 1177;
}
else
{
	vR[i] = 1176;
}

}

}

}
else
{
	if(input[i]>(1171+offset3))
{
	if(input[i]>(1173+offset3))
{
	if(input[i]==(1175+offset3))
{
	vR[i] = 1175;
}
else
{
	vR[i] = 1174;
}

}
else
{
	if(input[i]==(1173+offset3))
{
	vR[i] = 1173;
}
else
{
	vR[i] = 1172;
}

}

}
else
{
	if(input[i]>(1169+offset3))
{
	if(input[i]==(1171+offset3))
{
	vR[i] = 1171;
}
else
{
	vR[i] = 1170;
}

}
else
{
	if(input[i]==(1169+offset3))
{
	vR[i] = 1169;
}
else
{
	vR[i] = 1168;
}

}

}

}

}
else
{
	if(input[i]>(1159+offset3))
{
	if(input[i]>(1163+offset3))
{
	if(input[i]>(1165+offset3))
{
	if(input[i]==(1167+offset3))
{
	vR[i] = 1167;
}
else
{
	vR[i] = 1166;
}

}
else
{
	if(input[i]==(1165+offset3))
{
	vR[i] = 1165;
}
else
{
	vR[i] = 1164;
}

}

}
else
{
	if(input[i]>(1161+offset3))
{
	if(input[i]==(1163+offset3))
{
	vR[i] = 1163;
}
else
{
	vR[i] = 1162;
}

}
else
{
	if(input[i]==(1161+offset3))
{
	vR[i] = 1161;
}
else
{
	vR[i] = 1160;
}

}

}

}
else
{
	if(input[i]>(1155+offset3))
{
	if(input[i]>(1157+offset3))
{
	if(input[i]==(1159+offset3))
{
	vR[i] = 1159;
}
else
{
	vR[i] = 1158;
}

}
else
{
	if(input[i]==(1157+offset3))
{
	vR[i] = 1157;
}
else
{
	vR[i] = 1156;
}

}

}
else
{
	if(input[i]>(1153+offset3))
{
	if(input[i]==(1155+offset3))
{
	vR[i] = 1155;
}
else
{
	vR[i] = 1154;
}

}
else
{
	if(input[i]==(1153+offset3))
{
	vR[i] = 1153;
}
else
{
	vR[i] = 1152;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(1087+offset3))
{
	if(input[i]>(1119+offset3))
{
	if(input[i]>(1135+offset3))
{
	if(input[i]>(1143+offset3))
{
	if(input[i]>(1147+offset3))
{
	if(input[i]>(1149+offset3))
{
	if(input[i]==(1151+offset3))
{
	vR[i] = 1151;
}
else
{
	vR[i] = 1150;
}

}
else
{
	if(input[i]==(1149+offset3))
{
	vR[i] = 1149;
}
else
{
	vR[i] = 1148;
}

}

}
else
{
	if(input[i]>(1145+offset3))
{
	if(input[i]==(1147+offset3))
{
	vR[i] = 1147;
}
else
{
	vR[i] = 1146;
}

}
else
{
	if(input[i]==(1145+offset3))
{
	vR[i] = 1145;
}
else
{
	vR[i] = 1144;
}

}

}

}
else
{
	if(input[i]>(1139+offset3))
{
	if(input[i]>(1141+offset3))
{
	if(input[i]==(1143+offset3))
{
	vR[i] = 1143;
}
else
{
	vR[i] = 1142;
}

}
else
{
	if(input[i]==(1141+offset3))
{
	vR[i] = 1141;
}
else
{
	vR[i] = 1140;
}

}

}
else
{
	if(input[i]>(1137+offset3))
{
	if(input[i]==(1139+offset3))
{
	vR[i] = 1139;
}
else
{
	vR[i] = 1138;
}

}
else
{
	if(input[i]==(1137+offset3))
{
	vR[i] = 1137;
}
else
{
	vR[i] = 1136;
}

}

}

}

}
else
{
	if(input[i]>(1127+offset3))
{
	if(input[i]>(1131+offset3))
{
	if(input[i]>(1133+offset3))
{
	if(input[i]==(1135+offset3))
{
	vR[i] = 1135;
}
else
{
	vR[i] = 1134;
}

}
else
{
	if(input[i]==(1133+offset3))
{
	vR[i] = 1133;
}
else
{
	vR[i] = 1132;
}

}

}
else
{
	if(input[i]>(1129+offset3))
{
	if(input[i]==(1131+offset3))
{
	vR[i] = 1131;
}
else
{
	vR[i] = 1130;
}

}
else
{
	if(input[i]==(1129+offset3))
{
	vR[i] = 1129;
}
else
{
	vR[i] = 1128;
}

}

}

}
else
{
	if(input[i]>(1123+offset3))
{
	if(input[i]>(1125+offset3))
{
	if(input[i]==(1127+offset3))
{
	vR[i] = 1127;
}
else
{
	vR[i] = 1126;
}

}
else
{
	if(input[i]==(1125+offset3))
{
	vR[i] = 1125;
}
else
{
	vR[i] = 1124;
}

}

}
else
{
	if(input[i]>(1121+offset3))
{
	if(input[i]==(1123+offset3))
{
	vR[i] = 1123;
}
else
{
	vR[i] = 1122;
}

}
else
{
	if(input[i]==(1121+offset3))
{
	vR[i] = 1121;
}
else
{
	vR[i] = 1120;
}

}

}

}

}

}
else
{
	if(input[i]>(1103+offset3))
{
	if(input[i]>(1111+offset3))
{
	if(input[i]>(1115+offset3))
{
	if(input[i]>(1117+offset3))
{
	if(input[i]==(1119+offset3))
{
	vR[i] = 1119;
}
else
{
	vR[i] = 1118;
}

}
else
{
	if(input[i]==(1117+offset3))
{
	vR[i] = 1117;
}
else
{
	vR[i] = 1116;
}

}

}
else
{
	if(input[i]>(1113+offset3))
{
	if(input[i]==(1115+offset3))
{
	vR[i] = 1115;
}
else
{
	vR[i] = 1114;
}

}
else
{
	if(input[i]==(1113+offset3))
{
	vR[i] = 1113;
}
else
{
	vR[i] = 1112;
}

}

}

}
else
{
	if(input[i]>(1107+offset3))
{
	if(input[i]>(1109+offset3))
{
	if(input[i]==(1111+offset3))
{
	vR[i] = 1111;
}
else
{
	vR[i] = 1110;
}

}
else
{
	if(input[i]==(1109+offset3))
{
	vR[i] = 1109;
}
else
{
	vR[i] = 1108;
}

}

}
else
{
	if(input[i]>(1105+offset3))
{
	if(input[i]==(1107+offset3))
{
	vR[i] = 1107;
}
else
{
	vR[i] = 1106;
}

}
else
{
	if(input[i]==(1105+offset3))
{
	vR[i] = 1105;
}
else
{
	vR[i] = 1104;
}

}

}

}

}
else
{
	if(input[i]>(1095+offset3))
{
	if(input[i]>(1099+offset3))
{
	if(input[i]>(1101+offset3))
{
	if(input[i]==(1103+offset3))
{
	vR[i] = 1103;
}
else
{
	vR[i] = 1102;
}

}
else
{
	if(input[i]==(1101+offset3))
{
	vR[i] = 1101;
}
else
{
	vR[i] = 1100;
}

}

}
else
{
	if(input[i]>(1097+offset3))
{
	if(input[i]==(1099+offset3))
{
	vR[i] = 1099;
}
else
{
	vR[i] = 1098;
}

}
else
{
	if(input[i]==(1097+offset3))
{
	vR[i] = 1097;
}
else
{
	vR[i] = 1096;
}

}

}

}
else
{
	if(input[i]>(1091+offset3))
{
	if(input[i]>(1093+offset3))
{
	if(input[i]==(1095+offset3))
{
	vR[i] = 1095;
}
else
{
	vR[i] = 1094;
}

}
else
{
	if(input[i]==(1093+offset3))
{
	vR[i] = 1093;
}
else
{
	vR[i] = 1092;
}

}

}
else
{
	if(input[i]>(1089+offset3))
{
	if(input[i]==(1091+offset3))
{
	vR[i] = 1091;
}
else
{
	vR[i] = 1090;
}

}
else
{
	if(input[i]==(1089+offset3))
{
	vR[i] = 1089;
}
else
{
	vR[i] = 1088;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1055+offset3))
{
	if(input[i]>(1071+offset3))
{
	if(input[i]>(1079+offset3))
{
	if(input[i]>(1083+offset3))
{
	if(input[i]>(1085+offset3))
{
	if(input[i]==(1087+offset3))
{
	vR[i] = 1087;
}
else
{
	vR[i] = 1086;
}

}
else
{
	if(input[i]==(1085+offset3))
{
	vR[i] = 1085;
}
else
{
	vR[i] = 1084;
}

}

}
else
{
	if(input[i]>(1081+offset3))
{
	if(input[i]==(1083+offset3))
{
	vR[i] = 1083;
}
else
{
	vR[i] = 1082;
}

}
else
{
	if(input[i]==(1081+offset3))
{
	vR[i] = 1081;
}
else
{
	vR[i] = 1080;
}

}

}

}
else
{
	if(input[i]>(1075+offset3))
{
	if(input[i]>(1077+offset3))
{
	if(input[i]==(1079+offset3))
{
	vR[i] = 1079;
}
else
{
	vR[i] = 1078;
}

}
else
{
	if(input[i]==(1077+offset3))
{
	vR[i] = 1077;
}
else
{
	vR[i] = 1076;
}

}

}
else
{
	if(input[i]>(1073+offset3))
{
	if(input[i]==(1075+offset3))
{
	vR[i] = 1075;
}
else
{
	vR[i] = 1074;
}

}
else
{
	if(input[i]==(1073+offset3))
{
	vR[i] = 1073;
}
else
{
	vR[i] = 1072;
}

}

}

}

}
else
{
	if(input[i]>(1063+offset3))
{
	if(input[i]>(1067+offset3))
{
	if(input[i]>(1069+offset3))
{
	if(input[i]==(1071+offset3))
{
	vR[i] = 1071;
}
else
{
	vR[i] = 1070;
}

}
else
{
	if(input[i]==(1069+offset3))
{
	vR[i] = 1069;
}
else
{
	vR[i] = 1068;
}

}

}
else
{
	if(input[i]>(1065+offset3))
{
	if(input[i]==(1067+offset3))
{
	vR[i] = 1067;
}
else
{
	vR[i] = 1066;
}

}
else
{
	if(input[i]==(1065+offset3))
{
	vR[i] = 1065;
}
else
{
	vR[i] = 1064;
}

}

}

}
else
{
	if(input[i]>(1059+offset3))
{
	if(input[i]>(1061+offset3))
{
	if(input[i]==(1063+offset3))
{
	vR[i] = 1063;
}
else
{
	vR[i] = 1062;
}

}
else
{
	if(input[i]==(1061+offset3))
{
	vR[i] = 1061;
}
else
{
	vR[i] = 1060;
}

}

}
else
{
	if(input[i]>(1057+offset3))
{
	if(input[i]==(1059+offset3))
{
	vR[i] = 1059;
}
else
{
	vR[i] = 1058;
}

}
else
{
	if(input[i]==(1057+offset3))
{
	vR[i] = 1057;
}
else
{
	vR[i] = 1056;
}

}

}

}

}

}
else
{
	if(input[i]>(1039+offset3))
{
	if(input[i]>(1047+offset3))
{
	if(input[i]>(1051+offset3))
{
	if(input[i]>(1053+offset3))
{
	if(input[i]==(1055+offset3))
{
	vR[i] = 1055;
}
else
{
	vR[i] = 1054;
}

}
else
{
	if(input[i]==(1053+offset3))
{
	vR[i] = 1053;
}
else
{
	vR[i] = 1052;
}

}

}
else
{
	if(input[i]>(1049+offset3))
{
	if(input[i]==(1051+offset3))
{
	vR[i] = 1051;
}
else
{
	vR[i] = 1050;
}

}
else
{
	if(input[i]==(1049+offset3))
{
	vR[i] = 1049;
}
else
{
	vR[i] = 1048;
}

}

}

}
else
{
	if(input[i]>(1043+offset3))
{
	if(input[i]>(1045+offset3))
{
	if(input[i]==(1047+offset3))
{
	vR[i] = 1047;
}
else
{
	vR[i] = 1046;
}

}
else
{
	if(input[i]==(1045+offset3))
{
	vR[i] = 1045;
}
else
{
	vR[i] = 1044;
}

}

}
else
{
	if(input[i]>(1041+offset3))
{
	if(input[i]==(1043+offset3))
{
	vR[i] = 1043;
}
else
{
	vR[i] = 1042;
}

}
else
{
	if(input[i]==(1041+offset3))
{
	vR[i] = 1041;
}
else
{
	vR[i] = 1040;
}

}

}

}

}
else
{
	if(input[i]>(1031+offset3))
{
	if(input[i]>(1035+offset3))
{
	if(input[i]>(1037+offset3))
{
	if(input[i]==(1039+offset3))
{
	vR[i] = 1039;
}
else
{
	vR[i] = 1038;
}

}
else
{
	if(input[i]==(1037+offset3))
{
	vR[i] = 1037;
}
else
{
	vR[i] = 1036;
}

}

}
else
{
	if(input[i]>(1033+offset3))
{
	if(input[i]==(1035+offset3))
{
	vR[i] = 1035;
}
else
{
	vR[i] = 1034;
}

}
else
{
	if(input[i]==(1033+offset3))
{
	vR[i] = 1033;
}
else
{
	vR[i] = 1032;
}

}

}

}
else
{
	if(input[i]>(1027+offset3))
{
	if(input[i]>(1029+offset3))
{
	if(input[i]==(1031+offset3))
{
	vR[i] = 1031;
}
else
{
	vR[i] = 1030;
}

}
else
{
	if(input[i]==(1029+offset3))
{
	vR[i] = 1029;
}
else
{
	vR[i] = 1028;
}

}

}
else
{
	if(input[i]>(1025+offset3))
{
	if(input[i]==(1027+offset3))
{
	vR[i] = 1027;
}
else
{
	vR[i] = 1026;
}

}
else
{
	if(input[i]==(1025+offset3))
{
	vR[i] = 1025;
}
else
{
	vR[i] = 1024;
}

}

}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(511+offset3))
{
	if(input[i]>(767+offset3))
{
	if(input[i]>(895+offset3))
{
	if(input[i]>(959+offset3))
{
	if(input[i]>(991+offset3))
{
	if(input[i]>(1007+offset3))
{
	if(input[i]>(1015+offset3))
{
	if(input[i]>(1019+offset3))
{
	if(input[i]>(1021+offset3))
{
	if(input[i]==(1023+offset3))
{
	vR[i] = 1023;
}
else
{
	vR[i] = 1022;
}

}
else
{
	if(input[i]==(1021+offset3))
{
	vR[i] = 1021;
}
else
{
	vR[i] = 1020;
}

}

}
else
{
	if(input[i]>(1017+offset3))
{
	if(input[i]==(1019+offset3))
{
	vR[i] = 1019;
}
else
{
	vR[i] = 1018;
}

}
else
{
	if(input[i]==(1017+offset3))
{
	vR[i] = 1017;
}
else
{
	vR[i] = 1016;
}

}

}

}
else
{
	if(input[i]>(1011+offset3))
{
	if(input[i]>(1013+offset3))
{
	if(input[i]==(1015+offset3))
{
	vR[i] = 1015;
}
else
{
	vR[i] = 1014;
}

}
else
{
	if(input[i]==(1013+offset3))
{
	vR[i] = 1013;
}
else
{
	vR[i] = 1012;
}

}

}
else
{
	if(input[i]>(1009+offset3))
{
	if(input[i]==(1011+offset3))
{
	vR[i] = 1011;
}
else
{
	vR[i] = 1010;
}

}
else
{
	if(input[i]==(1009+offset3))
{
	vR[i] = 1009;
}
else
{
	vR[i] = 1008;
}

}

}

}

}
else
{
	if(input[i]>(999+offset3))
{
	if(input[i]>(1003+offset3))
{
	if(input[i]>(1005+offset3))
{
	if(input[i]==(1007+offset3))
{
	vR[i] = 1007;
}
else
{
	vR[i] = 1006;
}

}
else
{
	if(input[i]==(1005+offset3))
{
	vR[i] = 1005;
}
else
{
	vR[i] = 1004;
}

}

}
else
{
	if(input[i]>(1001+offset3))
{
	if(input[i]==(1003+offset3))
{
	vR[i] = 1003;
}
else
{
	vR[i] = 1002;
}

}
else
{
	if(input[i]==(1001+offset3))
{
	vR[i] = 1001;
}
else
{
	vR[i] = 1000;
}

}

}

}
else
{
	if(input[i]>(995+offset3))
{
	if(input[i]>(997+offset3))
{
	if(input[i]==(999+offset3))
{
	vR[i] = 999;
}
else
{
	vR[i] = 998;
}

}
else
{
	if(input[i]==(997+offset3))
{
	vR[i] = 997;
}
else
{
	vR[i] = 996;
}

}

}
else
{
	if(input[i]>(993+offset3))
{
	if(input[i]==(995+offset3))
{
	vR[i] = 995;
}
else
{
	vR[i] = 994;
}

}
else
{
	if(input[i]==(993+offset3))
{
	vR[i] = 993;
}
else
{
	vR[i] = 992;
}

}

}

}

}

}
else
{
	if(input[i]>(975+offset3))
{
	if(input[i]>(983+offset3))
{
	if(input[i]>(987+offset3))
{
	if(input[i]>(989+offset3))
{
	if(input[i]==(991+offset3))
{
	vR[i] = 991;
}
else
{
	vR[i] = 990;
}

}
else
{
	if(input[i]==(989+offset3))
{
	vR[i] = 989;
}
else
{
	vR[i] = 988;
}

}

}
else
{
	if(input[i]>(985+offset3))
{
	if(input[i]==(987+offset3))
{
	vR[i] = 987;
}
else
{
	vR[i] = 986;
}

}
else
{
	if(input[i]==(985+offset3))
{
	vR[i] = 985;
}
else
{
	vR[i] = 984;
}

}

}

}
else
{
	if(input[i]>(979+offset3))
{
	if(input[i]>(981+offset3))
{
	if(input[i]==(983+offset3))
{
	vR[i] = 983;
}
else
{
	vR[i] = 982;
}

}
else
{
	if(input[i]==(981+offset3))
{
	vR[i] = 981;
}
else
{
	vR[i] = 980;
}

}

}
else
{
	if(input[i]>(977+offset3))
{
	if(input[i]==(979+offset3))
{
	vR[i] = 979;
}
else
{
	vR[i] = 978;
}

}
else
{
	if(input[i]==(977+offset3))
{
	vR[i] = 977;
}
else
{
	vR[i] = 976;
}

}

}

}

}
else
{
	if(input[i]>(967+offset3))
{
	if(input[i]>(971+offset3))
{
	if(input[i]>(973+offset3))
{
	if(input[i]==(975+offset3))
{
	vR[i] = 975;
}
else
{
	vR[i] = 974;
}

}
else
{
	if(input[i]==(973+offset3))
{
	vR[i] = 973;
}
else
{
	vR[i] = 972;
}

}

}
else
{
	if(input[i]>(969+offset3))
{
	if(input[i]==(971+offset3))
{
	vR[i] = 971;
}
else
{
	vR[i] = 970;
}

}
else
{
	if(input[i]==(969+offset3))
{
	vR[i] = 969;
}
else
{
	vR[i] = 968;
}

}

}

}
else
{
	if(input[i]>(963+offset3))
{
	if(input[i]>(965+offset3))
{
	if(input[i]==(967+offset3))
{
	vR[i] = 967;
}
else
{
	vR[i] = 966;
}

}
else
{
	if(input[i]==(965+offset3))
{
	vR[i] = 965;
}
else
{
	vR[i] = 964;
}

}

}
else
{
	if(input[i]>(961+offset3))
{
	if(input[i]==(963+offset3))
{
	vR[i] = 963;
}
else
{
	vR[i] = 962;
}

}
else
{
	if(input[i]==(961+offset3))
{
	vR[i] = 961;
}
else
{
	vR[i] = 960;
}

}

}

}

}

}

}
else
{
	if(input[i]>(927+offset3))
{
	if(input[i]>(943+offset3))
{
	if(input[i]>(951+offset3))
{
	if(input[i]>(955+offset3))
{
	if(input[i]>(957+offset3))
{
	if(input[i]==(959+offset3))
{
	vR[i] = 959;
}
else
{
	vR[i] = 958;
}

}
else
{
	if(input[i]==(957+offset3))
{
	vR[i] = 957;
}
else
{
	vR[i] = 956;
}

}

}
else
{
	if(input[i]>(953+offset3))
{
	if(input[i]==(955+offset3))
{
	vR[i] = 955;
}
else
{
	vR[i] = 954;
}

}
else
{
	if(input[i]==(953+offset3))
{
	vR[i] = 953;
}
else
{
	vR[i] = 952;
}

}

}

}
else
{
	if(input[i]>(947+offset3))
{
	if(input[i]>(949+offset3))
{
	if(input[i]==(951+offset3))
{
	vR[i] = 951;
}
else
{
	vR[i] = 950;
}

}
else
{
	if(input[i]==(949+offset3))
{
	vR[i] = 949;
}
else
{
	vR[i] = 948;
}

}

}
else
{
	if(input[i]>(945+offset3))
{
	if(input[i]==(947+offset3))
{
	vR[i] = 947;
}
else
{
	vR[i] = 946;
}

}
else
{
	if(input[i]==(945+offset3))
{
	vR[i] = 945;
}
else
{
	vR[i] = 944;
}

}

}

}

}
else
{
	if(input[i]>(935+offset3))
{
	if(input[i]>(939+offset3))
{
	if(input[i]>(941+offset3))
{
	if(input[i]==(943+offset3))
{
	vR[i] = 943;
}
else
{
	vR[i] = 942;
}

}
else
{
	if(input[i]==(941+offset3))
{
	vR[i] = 941;
}
else
{
	vR[i] = 940;
}

}

}
else
{
	if(input[i]>(937+offset3))
{
	if(input[i]==(939+offset3))
{
	vR[i] = 939;
}
else
{
	vR[i] = 938;
}

}
else
{
	if(input[i]==(937+offset3))
{
	vR[i] = 937;
}
else
{
	vR[i] = 936;
}

}

}

}
else
{
	if(input[i]>(931+offset3))
{
	if(input[i]>(933+offset3))
{
	if(input[i]==(935+offset3))
{
	vR[i] = 935;
}
else
{
	vR[i] = 934;
}

}
else
{
	if(input[i]==(933+offset3))
{
	vR[i] = 933;
}
else
{
	vR[i] = 932;
}

}

}
else
{
	if(input[i]>(929+offset3))
{
	if(input[i]==(931+offset3))
{
	vR[i] = 931;
}
else
{
	vR[i] = 930;
}

}
else
{
	if(input[i]==(929+offset3))
{
	vR[i] = 929;
}
else
{
	vR[i] = 928;
}

}

}

}

}

}
else
{
	if(input[i]>(911+offset3))
{
	if(input[i]>(919+offset3))
{
	if(input[i]>(923+offset3))
{
	if(input[i]>(925+offset3))
{
	if(input[i]==(927+offset3))
{
	vR[i] = 927;
}
else
{
	vR[i] = 926;
}

}
else
{
	if(input[i]==(925+offset3))
{
	vR[i] = 925;
}
else
{
	vR[i] = 924;
}

}

}
else
{
	if(input[i]>(921+offset3))
{
	if(input[i]==(923+offset3))
{
	vR[i] = 923;
}
else
{
	vR[i] = 922;
}

}
else
{
	if(input[i]==(921+offset3))
{
	vR[i] = 921;
}
else
{
	vR[i] = 920;
}

}

}

}
else
{
	if(input[i]>(915+offset3))
{
	if(input[i]>(917+offset3))
{
	if(input[i]==(919+offset3))
{
	vR[i] = 919;
}
else
{
	vR[i] = 918;
}

}
else
{
	if(input[i]==(917+offset3))
{
	vR[i] = 917;
}
else
{
	vR[i] = 916;
}

}

}
else
{
	if(input[i]>(913+offset3))
{
	if(input[i]==(915+offset3))
{
	vR[i] = 915;
}
else
{
	vR[i] = 914;
}

}
else
{
	if(input[i]==(913+offset3))
{
	vR[i] = 913;
}
else
{
	vR[i] = 912;
}

}

}

}

}
else
{
	if(input[i]>(903+offset3))
{
	if(input[i]>(907+offset3))
{
	if(input[i]>(909+offset3))
{
	if(input[i]==(911+offset3))
{
	vR[i] = 911;
}
else
{
	vR[i] = 910;
}

}
else
{
	if(input[i]==(909+offset3))
{
	vR[i] = 909;
}
else
{
	vR[i] = 908;
}

}

}
else
{
	if(input[i]>(905+offset3))
{
	if(input[i]==(907+offset3))
{
	vR[i] = 907;
}
else
{
	vR[i] = 906;
}

}
else
{
	if(input[i]==(905+offset3))
{
	vR[i] = 905;
}
else
{
	vR[i] = 904;
}

}

}

}
else
{
	if(input[i]>(899+offset3))
{
	if(input[i]>(901+offset3))
{
	if(input[i]==(903+offset3))
{
	vR[i] = 903;
}
else
{
	vR[i] = 902;
}

}
else
{
	if(input[i]==(901+offset3))
{
	vR[i] = 901;
}
else
{
	vR[i] = 900;
}

}

}
else
{
	if(input[i]>(897+offset3))
{
	if(input[i]==(899+offset3))
{
	vR[i] = 899;
}
else
{
	vR[i] = 898;
}

}
else
{
	if(input[i]==(897+offset3))
{
	vR[i] = 897;
}
else
{
	vR[i] = 896;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(831+offset3))
{
	if(input[i]>(863+offset3))
{
	if(input[i]>(879+offset3))
{
	if(input[i]>(887+offset3))
{
	if(input[i]>(891+offset3))
{
	if(input[i]>(893+offset3))
{
	if(input[i]==(895+offset3))
{
	vR[i] = 895;
}
else
{
	vR[i] = 894;
}

}
else
{
	if(input[i]==(893+offset3))
{
	vR[i] = 893;
}
else
{
	vR[i] = 892;
}

}

}
else
{
	if(input[i]>(889+offset3))
{
	if(input[i]==(891+offset3))
{
	vR[i] = 891;
}
else
{
	vR[i] = 890;
}

}
else
{
	if(input[i]==(889+offset3))
{
	vR[i] = 889;
}
else
{
	vR[i] = 888;
}

}

}

}
else
{
	if(input[i]>(883+offset3))
{
	if(input[i]>(885+offset3))
{
	if(input[i]==(887+offset3))
{
	vR[i] = 887;
}
else
{
	vR[i] = 886;
}

}
else
{
	if(input[i]==(885+offset3))
{
	vR[i] = 885;
}
else
{
	vR[i] = 884;
}

}

}
else
{
	if(input[i]>(881+offset3))
{
	if(input[i]==(883+offset3))
{
	vR[i] = 883;
}
else
{
	vR[i] = 882;
}

}
else
{
	if(input[i]==(881+offset3))
{
	vR[i] = 881;
}
else
{
	vR[i] = 880;
}

}

}

}

}
else
{
	if(input[i]>(871+offset3))
{
	if(input[i]>(875+offset3))
{
	if(input[i]>(877+offset3))
{
	if(input[i]==(879+offset3))
{
	vR[i] = 879;
}
else
{
	vR[i] = 878;
}

}
else
{
	if(input[i]==(877+offset3))
{
	vR[i] = 877;
}
else
{
	vR[i] = 876;
}

}

}
else
{
	if(input[i]>(873+offset3))
{
	if(input[i]==(875+offset3))
{
	vR[i] = 875;
}
else
{
	vR[i] = 874;
}

}
else
{
	if(input[i]==(873+offset3))
{
	vR[i] = 873;
}
else
{
	vR[i] = 872;
}

}

}

}
else
{
	if(input[i]>(867+offset3))
{
	if(input[i]>(869+offset3))
{
	if(input[i]==(871+offset3))
{
	vR[i] = 871;
}
else
{
	vR[i] = 870;
}

}
else
{
	if(input[i]==(869+offset3))
{
	vR[i] = 869;
}
else
{
	vR[i] = 868;
}

}

}
else
{
	if(input[i]>(865+offset3))
{
	if(input[i]==(867+offset3))
{
	vR[i] = 867;
}
else
{
	vR[i] = 866;
}

}
else
{
	if(input[i]==(865+offset3))
{
	vR[i] = 865;
}
else
{
	vR[i] = 864;
}

}

}

}

}

}
else
{
	if(input[i]>(847+offset3))
{
	if(input[i]>(855+offset3))
{
	if(input[i]>(859+offset3))
{
	if(input[i]>(861+offset3))
{
	if(input[i]==(863+offset3))
{
	vR[i] = 863;
}
else
{
	vR[i] = 862;
}

}
else
{
	if(input[i]==(861+offset3))
{
	vR[i] = 861;
}
else
{
	vR[i] = 860;
}

}

}
else
{
	if(input[i]>(857+offset3))
{
	if(input[i]==(859+offset3))
{
	vR[i] = 859;
}
else
{
	vR[i] = 858;
}

}
else
{
	if(input[i]==(857+offset3))
{
	vR[i] = 857;
}
else
{
	vR[i] = 856;
}

}

}

}
else
{
	if(input[i]>(851+offset3))
{
	if(input[i]>(853+offset3))
{
	if(input[i]==(855+offset3))
{
	vR[i] = 855;
}
else
{
	vR[i] = 854;
}

}
else
{
	if(input[i]==(853+offset3))
{
	vR[i] = 853;
}
else
{
	vR[i] = 852;
}

}

}
else
{
	if(input[i]>(849+offset3))
{
	if(input[i]==(851+offset3))
{
	vR[i] = 851;
}
else
{
	vR[i] = 850;
}

}
else
{
	if(input[i]==(849+offset3))
{
	vR[i] = 849;
}
else
{
	vR[i] = 848;
}

}

}

}

}
else
{
	if(input[i]>(839+offset3))
{
	if(input[i]>(843+offset3))
{
	if(input[i]>(845+offset3))
{
	if(input[i]==(847+offset3))
{
	vR[i] = 847;
}
else
{
	vR[i] = 846;
}

}
else
{
	if(input[i]==(845+offset3))
{
	vR[i] = 845;
}
else
{
	vR[i] = 844;
}

}

}
else
{
	if(input[i]>(841+offset3))
{
	if(input[i]==(843+offset3))
{
	vR[i] = 843;
}
else
{
	vR[i] = 842;
}

}
else
{
	if(input[i]==(841+offset3))
{
	vR[i] = 841;
}
else
{
	vR[i] = 840;
}

}

}

}
else
{
	if(input[i]>(835+offset3))
{
	if(input[i]>(837+offset3))
{
	if(input[i]==(839+offset3))
{
	vR[i] = 839;
}
else
{
	vR[i] = 838;
}

}
else
{
	if(input[i]==(837+offset3))
{
	vR[i] = 837;
}
else
{
	vR[i] = 836;
}

}

}
else
{
	if(input[i]>(833+offset3))
{
	if(input[i]==(835+offset3))
{
	vR[i] = 835;
}
else
{
	vR[i] = 834;
}

}
else
{
	if(input[i]==(833+offset3))
{
	vR[i] = 833;
}
else
{
	vR[i] = 832;
}

}

}

}

}

}

}
else
{
	if(input[i]>(799+offset3))
{
	if(input[i]>(815+offset3))
{
	if(input[i]>(823+offset3))
{
	if(input[i]>(827+offset3))
{
	if(input[i]>(829+offset3))
{
	if(input[i]==(831+offset3))
{
	vR[i] = 831;
}
else
{
	vR[i] = 830;
}

}
else
{
	if(input[i]==(829+offset3))
{
	vR[i] = 829;
}
else
{
	vR[i] = 828;
}

}

}
else
{
	if(input[i]>(825+offset3))
{
	if(input[i]==(827+offset3))
{
	vR[i] = 827;
}
else
{
	vR[i] = 826;
}

}
else
{
	if(input[i]==(825+offset3))
{
	vR[i] = 825;
}
else
{
	vR[i] = 824;
}

}

}

}
else
{
	if(input[i]>(819+offset3))
{
	if(input[i]>(821+offset3))
{
	if(input[i]==(823+offset3))
{
	vR[i] = 823;
}
else
{
	vR[i] = 822;
}

}
else
{
	if(input[i]==(821+offset3))
{
	vR[i] = 821;
}
else
{
	vR[i] = 820;
}

}

}
else
{
	if(input[i]>(817+offset3))
{
	if(input[i]==(819+offset3))
{
	vR[i] = 819;
}
else
{
	vR[i] = 818;
}

}
else
{
	if(input[i]==(817+offset3))
{
	vR[i] = 817;
}
else
{
	vR[i] = 816;
}

}

}

}

}
else
{
	if(input[i]>(807+offset3))
{
	if(input[i]>(811+offset3))
{
	if(input[i]>(813+offset3))
{
	if(input[i]==(815+offset3))
{
	vR[i] = 815;
}
else
{
	vR[i] = 814;
}

}
else
{
	if(input[i]==(813+offset3))
{
	vR[i] = 813;
}
else
{
	vR[i] = 812;
}

}

}
else
{
	if(input[i]>(809+offset3))
{
	if(input[i]==(811+offset3))
{
	vR[i] = 811;
}
else
{
	vR[i] = 810;
}

}
else
{
	if(input[i]==(809+offset3))
{
	vR[i] = 809;
}
else
{
	vR[i] = 808;
}

}

}

}
else
{
	if(input[i]>(803+offset3))
{
	if(input[i]>(805+offset3))
{
	if(input[i]==(807+offset3))
{
	vR[i] = 807;
}
else
{
	vR[i] = 806;
}

}
else
{
	if(input[i]==(805+offset3))
{
	vR[i] = 805;
}
else
{
	vR[i] = 804;
}

}

}
else
{
	if(input[i]>(801+offset3))
{
	if(input[i]==(803+offset3))
{
	vR[i] = 803;
}
else
{
	vR[i] = 802;
}

}
else
{
	if(input[i]==(801+offset3))
{
	vR[i] = 801;
}
else
{
	vR[i] = 800;
}

}

}

}

}

}
else
{
	if(input[i]>(783+offset3))
{
	if(input[i]>(791+offset3))
{
	if(input[i]>(795+offset3))
{
	if(input[i]>(797+offset3))
{
	if(input[i]==(799+offset3))
{
	vR[i] = 799;
}
else
{
	vR[i] = 798;
}

}
else
{
	if(input[i]==(797+offset3))
{
	vR[i] = 797;
}
else
{
	vR[i] = 796;
}

}

}
else
{
	if(input[i]>(793+offset3))
{
	if(input[i]==(795+offset3))
{
	vR[i] = 795;
}
else
{
	vR[i] = 794;
}

}
else
{
	if(input[i]==(793+offset3))
{
	vR[i] = 793;
}
else
{
	vR[i] = 792;
}

}

}

}
else
{
	if(input[i]>(787+offset3))
{
	if(input[i]>(789+offset3))
{
	if(input[i]==(791+offset3))
{
	vR[i] = 791;
}
else
{
	vR[i] = 790;
}

}
else
{
	if(input[i]==(789+offset3))
{
	vR[i] = 789;
}
else
{
	vR[i] = 788;
}

}

}
else
{
	if(input[i]>(785+offset3))
{
	if(input[i]==(787+offset3))
{
	vR[i] = 787;
}
else
{
	vR[i] = 786;
}

}
else
{
	if(input[i]==(785+offset3))
{
	vR[i] = 785;
}
else
{
	vR[i] = 784;
}

}

}

}

}
else
{
	if(input[i]>(775+offset3))
{
	if(input[i]>(779+offset3))
{
	if(input[i]>(781+offset3))
{
	if(input[i]==(783+offset3))
{
	vR[i] = 783;
}
else
{
	vR[i] = 782;
}

}
else
{
	if(input[i]==(781+offset3))
{
	vR[i] = 781;
}
else
{
	vR[i] = 780;
}

}

}
else
{
	if(input[i]>(777+offset3))
{
	if(input[i]==(779+offset3))
{
	vR[i] = 779;
}
else
{
	vR[i] = 778;
}

}
else
{
	if(input[i]==(777+offset3))
{
	vR[i] = 777;
}
else
{
	vR[i] = 776;
}

}

}

}
else
{
	if(input[i]>(771+offset3))
{
	if(input[i]>(773+offset3))
{
	if(input[i]==(775+offset3))
{
	vR[i] = 775;
}
else
{
	vR[i] = 774;
}

}
else
{
	if(input[i]==(773+offset3))
{
	vR[i] = 773;
}
else
{
	vR[i] = 772;
}

}

}
else
{
	if(input[i]>(769+offset3))
{
	if(input[i]==(771+offset3))
{
	vR[i] = 771;
}
else
{
	vR[i] = 770;
}

}
else
{
	if(input[i]==(769+offset3))
{
	vR[i] = 769;
}
else
{
	vR[i] = 768;
}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(639+offset3))
{
	if(input[i]>(703+offset3))
{
	if(input[i]>(735+offset3))
{
	if(input[i]>(751+offset3))
{
	if(input[i]>(759+offset3))
{
	if(input[i]>(763+offset3))
{
	if(input[i]>(765+offset3))
{
	if(input[i]==(767+offset3))
{
	vR[i] = 767;
}
else
{
	vR[i] = 766;
}

}
else
{
	if(input[i]==(765+offset3))
{
	vR[i] = 765;
}
else
{
	vR[i] = 764;
}

}

}
else
{
	if(input[i]>(761+offset3))
{
	if(input[i]==(763+offset3))
{
	vR[i] = 763;
}
else
{
	vR[i] = 762;
}

}
else
{
	if(input[i]==(761+offset3))
{
	vR[i] = 761;
}
else
{
	vR[i] = 760;
}

}

}

}
else
{
	if(input[i]>(755+offset3))
{
	if(input[i]>(757+offset3))
{
	if(input[i]==(759+offset3))
{
	vR[i] = 759;
}
else
{
	vR[i] = 758;
}

}
else
{
	if(input[i]==(757+offset3))
{
	vR[i] = 757;
}
else
{
	vR[i] = 756;
}

}

}
else
{
	if(input[i]>(753+offset3))
{
	if(input[i]==(755+offset3))
{
	vR[i] = 755;
}
else
{
	vR[i] = 754;
}

}
else
{
	if(input[i]==(753+offset3))
{
	vR[i] = 753;
}
else
{
	vR[i] = 752;
}

}

}

}

}
else
{
	if(input[i]>(743+offset3))
{
	if(input[i]>(747+offset3))
{
	if(input[i]>(749+offset3))
{
	if(input[i]==(751+offset3))
{
	vR[i] = 751;
}
else
{
	vR[i] = 750;
}

}
else
{
	if(input[i]==(749+offset3))
{
	vR[i] = 749;
}
else
{
	vR[i] = 748;
}

}

}
else
{
	if(input[i]>(745+offset3))
{
	if(input[i]==(747+offset3))
{
	vR[i] = 747;
}
else
{
	vR[i] = 746;
}

}
else
{
	if(input[i]==(745+offset3))
{
	vR[i] = 745;
}
else
{
	vR[i] = 744;
}

}

}

}
else
{
	if(input[i]>(739+offset3))
{
	if(input[i]>(741+offset3))
{
	if(input[i]==(743+offset3))
{
	vR[i] = 743;
}
else
{
	vR[i] = 742;
}

}
else
{
	if(input[i]==(741+offset3))
{
	vR[i] = 741;
}
else
{
	vR[i] = 740;
}

}

}
else
{
	if(input[i]>(737+offset3))
{
	if(input[i]==(739+offset3))
{
	vR[i] = 739;
}
else
{
	vR[i] = 738;
}

}
else
{
	if(input[i]==(737+offset3))
{
	vR[i] = 737;
}
else
{
	vR[i] = 736;
}

}

}

}

}

}
else
{
	if(input[i]>(719+offset3))
{
	if(input[i]>(727+offset3))
{
	if(input[i]>(731+offset3))
{
	if(input[i]>(733+offset3))
{
	if(input[i]==(735+offset3))
{
	vR[i] = 735;
}
else
{
	vR[i] = 734;
}

}
else
{
	if(input[i]==(733+offset3))
{
	vR[i] = 733;
}
else
{
	vR[i] = 732;
}

}

}
else
{
	if(input[i]>(729+offset3))
{
	if(input[i]==(731+offset3))
{
	vR[i] = 731;
}
else
{
	vR[i] = 730;
}

}
else
{
	if(input[i]==(729+offset3))
{
	vR[i] = 729;
}
else
{
	vR[i] = 728;
}

}

}

}
else
{
	if(input[i]>(723+offset3))
{
	if(input[i]>(725+offset3))
{
	if(input[i]==(727+offset3))
{
	vR[i] = 727;
}
else
{
	vR[i] = 726;
}

}
else
{
	if(input[i]==(725+offset3))
{
	vR[i] = 725;
}
else
{
	vR[i] = 724;
}

}

}
else
{
	if(input[i]>(721+offset3))
{
	if(input[i]==(723+offset3))
{
	vR[i] = 723;
}
else
{
	vR[i] = 722;
}

}
else
{
	if(input[i]==(721+offset3))
{
	vR[i] = 721;
}
else
{
	vR[i] = 720;
}

}

}

}

}
else
{
	if(input[i]>(711+offset3))
{
	if(input[i]>(715+offset3))
{
	if(input[i]>(717+offset3))
{
	if(input[i]==(719+offset3))
{
	vR[i] = 719;
}
else
{
	vR[i] = 718;
}

}
else
{
	if(input[i]==(717+offset3))
{
	vR[i] = 717;
}
else
{
	vR[i] = 716;
}

}

}
else
{
	if(input[i]>(713+offset3))
{
	if(input[i]==(715+offset3))
{
	vR[i] = 715;
}
else
{
	vR[i] = 714;
}

}
else
{
	if(input[i]==(713+offset3))
{
	vR[i] = 713;
}
else
{
	vR[i] = 712;
}

}

}

}
else
{
	if(input[i]>(707+offset3))
{
	if(input[i]>(709+offset3))
{
	if(input[i]==(711+offset3))
{
	vR[i] = 711;
}
else
{
	vR[i] = 710;
}

}
else
{
	if(input[i]==(709+offset3))
{
	vR[i] = 709;
}
else
{
	vR[i] = 708;
}

}

}
else
{
	if(input[i]>(705+offset3))
{
	if(input[i]==(707+offset3))
{
	vR[i] = 707;
}
else
{
	vR[i] = 706;
}

}
else
{
	if(input[i]==(705+offset3))
{
	vR[i] = 705;
}
else
{
	vR[i] = 704;
}

}

}

}

}

}

}
else
{
	if(input[i]>(671+offset3))
{
	if(input[i]>(687+offset3))
{
	if(input[i]>(695+offset3))
{
	if(input[i]>(699+offset3))
{
	if(input[i]>(701+offset3))
{
	if(input[i]==(703+offset3))
{
	vR[i] = 703;
}
else
{
	vR[i] = 702;
}

}
else
{
	if(input[i]==(701+offset3))
{
	vR[i] = 701;
}
else
{
	vR[i] = 700;
}

}

}
else
{
	if(input[i]>(697+offset3))
{
	if(input[i]==(699+offset3))
{
	vR[i] = 699;
}
else
{
	vR[i] = 698;
}

}
else
{
	if(input[i]==(697+offset3))
{
	vR[i] = 697;
}
else
{
	vR[i] = 696;
}

}

}

}
else
{
	if(input[i]>(691+offset3))
{
	if(input[i]>(693+offset3))
{
	if(input[i]==(695+offset3))
{
	vR[i] = 695;
}
else
{
	vR[i] = 694;
}

}
else
{
	if(input[i]==(693+offset3))
{
	vR[i] = 693;
}
else
{
	vR[i] = 692;
}

}

}
else
{
	if(input[i]>(689+offset3))
{
	if(input[i]==(691+offset3))
{
	vR[i] = 691;
}
else
{
	vR[i] = 690;
}

}
else
{
	if(input[i]==(689+offset3))
{
	vR[i] = 689;
}
else
{
	vR[i] = 688;
}

}

}

}

}
else
{
	if(input[i]>(679+offset3))
{
	if(input[i]>(683+offset3))
{
	if(input[i]>(685+offset3))
{
	if(input[i]==(687+offset3))
{
	vR[i] = 687;
}
else
{
	vR[i] = 686;
}

}
else
{
	if(input[i]==(685+offset3))
{
	vR[i] = 685;
}
else
{
	vR[i] = 684;
}

}

}
else
{
	if(input[i]>(681+offset3))
{
	if(input[i]==(683+offset3))
{
	vR[i] = 683;
}
else
{
	vR[i] = 682;
}

}
else
{
	if(input[i]==(681+offset3))
{
	vR[i] = 681;
}
else
{
	vR[i] = 680;
}

}

}

}
else
{
	if(input[i]>(675+offset3))
{
	if(input[i]>(677+offset3))
{
	if(input[i]==(679+offset3))
{
	vR[i] = 679;
}
else
{
	vR[i] = 678;
}

}
else
{
	if(input[i]==(677+offset3))
{
	vR[i] = 677;
}
else
{
	vR[i] = 676;
}

}

}
else
{
	if(input[i]>(673+offset3))
{
	if(input[i]==(675+offset3))
{
	vR[i] = 675;
}
else
{
	vR[i] = 674;
}

}
else
{
	if(input[i]==(673+offset3))
{
	vR[i] = 673;
}
else
{
	vR[i] = 672;
}

}

}

}

}

}
else
{
	if(input[i]>(655+offset3))
{
	if(input[i]>(663+offset3))
{
	if(input[i]>(667+offset3))
{
	if(input[i]>(669+offset3))
{
	if(input[i]==(671+offset3))
{
	vR[i] = 671;
}
else
{
	vR[i] = 670;
}

}
else
{
	if(input[i]==(669+offset3))
{
	vR[i] = 669;
}
else
{
	vR[i] = 668;
}

}

}
else
{
	if(input[i]>(665+offset3))
{
	if(input[i]==(667+offset3))
{
	vR[i] = 667;
}
else
{
	vR[i] = 666;
}

}
else
{
	if(input[i]==(665+offset3))
{
	vR[i] = 665;
}
else
{
	vR[i] = 664;
}

}

}

}
else
{
	if(input[i]>(659+offset3))
{
	if(input[i]>(661+offset3))
{
	if(input[i]==(663+offset3))
{
	vR[i] = 663;
}
else
{
	vR[i] = 662;
}

}
else
{
	if(input[i]==(661+offset3))
{
	vR[i] = 661;
}
else
{
	vR[i] = 660;
}

}

}
else
{
	if(input[i]>(657+offset3))
{
	if(input[i]==(659+offset3))
{
	vR[i] = 659;
}
else
{
	vR[i] = 658;
}

}
else
{
	if(input[i]==(657+offset3))
{
	vR[i] = 657;
}
else
{
	vR[i] = 656;
}

}

}

}

}
else
{
	if(input[i]>(647+offset3))
{
	if(input[i]>(651+offset3))
{
	if(input[i]>(653+offset3))
{
	if(input[i]==(655+offset3))
{
	vR[i] = 655;
}
else
{
	vR[i] = 654;
}

}
else
{
	if(input[i]==(653+offset3))
{
	vR[i] = 653;
}
else
{
	vR[i] = 652;
}

}

}
else
{
	if(input[i]>(649+offset3))
{
	if(input[i]==(651+offset3))
{
	vR[i] = 651;
}
else
{
	vR[i] = 650;
}

}
else
{
	if(input[i]==(649+offset3))
{
	vR[i] = 649;
}
else
{
	vR[i] = 648;
}

}

}

}
else
{
	if(input[i]>(643+offset3))
{
	if(input[i]>(645+offset3))
{
	if(input[i]==(647+offset3))
{
	vR[i] = 647;
}
else
{
	vR[i] = 646;
}

}
else
{
	if(input[i]==(645+offset3))
{
	vR[i] = 645;
}
else
{
	vR[i] = 644;
}

}

}
else
{
	if(input[i]>(641+offset3))
{
	if(input[i]==(643+offset3))
{
	vR[i] = 643;
}
else
{
	vR[i] = 642;
}

}
else
{
	if(input[i]==(641+offset3))
{
	vR[i] = 641;
}
else
{
	vR[i] = 640;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(575+offset3))
{
	if(input[i]>(607+offset3))
{
	if(input[i]>(623+offset3))
{
	if(input[i]>(631+offset3))
{
	if(input[i]>(635+offset3))
{
	if(input[i]>(637+offset3))
{
	if(input[i]==(639+offset3))
{
	vR[i] = 639;
}
else
{
	vR[i] = 638;
}

}
else
{
	if(input[i]==(637+offset3))
{
	vR[i] = 637;
}
else
{
	vR[i] = 636;
}

}

}
else
{
	if(input[i]>(633+offset3))
{
	if(input[i]==(635+offset3))
{
	vR[i] = 635;
}
else
{
	vR[i] = 634;
}

}
else
{
	if(input[i]==(633+offset3))
{
	vR[i] = 633;
}
else
{
	vR[i] = 632;
}

}

}

}
else
{
	if(input[i]>(627+offset3))
{
	if(input[i]>(629+offset3))
{
	if(input[i]==(631+offset3))
{
	vR[i] = 631;
}
else
{
	vR[i] = 630;
}

}
else
{
	if(input[i]==(629+offset3))
{
	vR[i] = 629;
}
else
{
	vR[i] = 628;
}

}

}
else
{
	if(input[i]>(625+offset3))
{
	if(input[i]==(627+offset3))
{
	vR[i] = 627;
}
else
{
	vR[i] = 626;
}

}
else
{
	if(input[i]==(625+offset3))
{
	vR[i] = 625;
}
else
{
	vR[i] = 624;
}

}

}

}

}
else
{
	if(input[i]>(615+offset3))
{
	if(input[i]>(619+offset3))
{
	if(input[i]>(621+offset3))
{
	if(input[i]==(623+offset3))
{
	vR[i] = 623;
}
else
{
	vR[i] = 622;
}

}
else
{
	if(input[i]==(621+offset3))
{
	vR[i] = 621;
}
else
{
	vR[i] = 620;
}

}

}
else
{
	if(input[i]>(617+offset3))
{
	if(input[i]==(619+offset3))
{
	vR[i] = 619;
}
else
{
	vR[i] = 618;
}

}
else
{
	if(input[i]==(617+offset3))
{
	vR[i] = 617;
}
else
{
	vR[i] = 616;
}

}

}

}
else
{
	if(input[i]>(611+offset3))
{
	if(input[i]>(613+offset3))
{
	if(input[i]==(615+offset3))
{
	vR[i] = 615;
}
else
{
	vR[i] = 614;
}

}
else
{
	if(input[i]==(613+offset3))
{
	vR[i] = 613;
}
else
{
	vR[i] = 612;
}

}

}
else
{
	if(input[i]>(609+offset3))
{
	if(input[i]==(611+offset3))
{
	vR[i] = 611;
}
else
{
	vR[i] = 610;
}

}
else
{
	if(input[i]==(609+offset3))
{
	vR[i] = 609;
}
else
{
	vR[i] = 608;
}

}

}

}

}

}
else
{
	if(input[i]>(591+offset3))
{
	if(input[i]>(599+offset3))
{
	if(input[i]>(603+offset3))
{
	if(input[i]>(605+offset3))
{
	if(input[i]==(607+offset3))
{
	vR[i] = 607;
}
else
{
	vR[i] = 606;
}

}
else
{
	if(input[i]==(605+offset3))
{
	vR[i] = 605;
}
else
{
	vR[i] = 604;
}

}

}
else
{
	if(input[i]>(601+offset3))
{
	if(input[i]==(603+offset3))
{
	vR[i] = 603;
}
else
{
	vR[i] = 602;
}

}
else
{
	if(input[i]==(601+offset3))
{
	vR[i] = 601;
}
else
{
	vR[i] = 600;
}

}

}

}
else
{
	if(input[i]>(595+offset3))
{
	if(input[i]>(597+offset3))
{
	if(input[i]==(599+offset3))
{
	vR[i] = 599;
}
else
{
	vR[i] = 598;
}

}
else
{
	if(input[i]==(597+offset3))
{
	vR[i] = 597;
}
else
{
	vR[i] = 596;
}

}

}
else
{
	if(input[i]>(593+offset3))
{
	if(input[i]==(595+offset3))
{
	vR[i] = 595;
}
else
{
	vR[i] = 594;
}

}
else
{
	if(input[i]==(593+offset3))
{
	vR[i] = 593;
}
else
{
	vR[i] = 592;
}

}

}

}

}
else
{
	if(input[i]>(583+offset3))
{
	if(input[i]>(587+offset3))
{
	if(input[i]>(589+offset3))
{
	if(input[i]==(591+offset3))
{
	vR[i] = 591;
}
else
{
	vR[i] = 590;
}

}
else
{
	if(input[i]==(589+offset3))
{
	vR[i] = 589;
}
else
{
	vR[i] = 588;
}

}

}
else
{
	if(input[i]>(585+offset3))
{
	if(input[i]==(587+offset3))
{
	vR[i] = 587;
}
else
{
	vR[i] = 586;
}

}
else
{
	if(input[i]==(585+offset3))
{
	vR[i] = 585;
}
else
{
	vR[i] = 584;
}

}

}

}
else
{
	if(input[i]>(579+offset3))
{
	if(input[i]>(581+offset3))
{
	if(input[i]==(583+offset3))
{
	vR[i] = 583;
}
else
{
	vR[i] = 582;
}

}
else
{
	if(input[i]==(581+offset3))
{
	vR[i] = 581;
}
else
{
	vR[i] = 580;
}

}

}
else
{
	if(input[i]>(577+offset3))
{
	if(input[i]==(579+offset3))
{
	vR[i] = 579;
}
else
{
	vR[i] = 578;
}

}
else
{
	if(input[i]==(577+offset3))
{
	vR[i] = 577;
}
else
{
	vR[i] = 576;
}

}

}

}

}

}

}
else
{
	if(input[i]>(543+offset3))
{
	if(input[i]>(559+offset3))
{
	if(input[i]>(567+offset3))
{
	if(input[i]>(571+offset3))
{
	if(input[i]>(573+offset3))
{
	if(input[i]==(575+offset3))
{
	vR[i] = 575;
}
else
{
	vR[i] = 574;
}

}
else
{
	if(input[i]==(573+offset3))
{
	vR[i] = 573;
}
else
{
	vR[i] = 572;
}

}

}
else
{
	if(input[i]>(569+offset3))
{
	if(input[i]==(571+offset3))
{
	vR[i] = 571;
}
else
{
	vR[i] = 570;
}

}
else
{
	if(input[i]==(569+offset3))
{
	vR[i] = 569;
}
else
{
	vR[i] = 568;
}

}

}

}
else
{
	if(input[i]>(563+offset3))
{
	if(input[i]>(565+offset3))
{
	if(input[i]==(567+offset3))
{
	vR[i] = 567;
}
else
{
	vR[i] = 566;
}

}
else
{
	if(input[i]==(565+offset3))
{
	vR[i] = 565;
}
else
{
	vR[i] = 564;
}

}

}
else
{
	if(input[i]>(561+offset3))
{
	if(input[i]==(563+offset3))
{
	vR[i] = 563;
}
else
{
	vR[i] = 562;
}

}
else
{
	if(input[i]==(561+offset3))
{
	vR[i] = 561;
}
else
{
	vR[i] = 560;
}

}

}

}

}
else
{
	if(input[i]>(551+offset3))
{
	if(input[i]>(555+offset3))
{
	if(input[i]>(557+offset3))
{
	if(input[i]==(559+offset3))
{
	vR[i] = 559;
}
else
{
	vR[i] = 558;
}

}
else
{
	if(input[i]==(557+offset3))
{
	vR[i] = 557;
}
else
{
	vR[i] = 556;
}

}

}
else
{
	if(input[i]>(553+offset3))
{
	if(input[i]==(555+offset3))
{
	vR[i] = 555;
}
else
{
	vR[i] = 554;
}

}
else
{
	if(input[i]==(553+offset3))
{
	vR[i] = 553;
}
else
{
	vR[i] = 552;
}

}

}

}
else
{
	if(input[i]>(547+offset3))
{
	if(input[i]>(549+offset3))
{
	if(input[i]==(551+offset3))
{
	vR[i] = 551;
}
else
{
	vR[i] = 550;
}

}
else
{
	if(input[i]==(549+offset3))
{
	vR[i] = 549;
}
else
{
	vR[i] = 548;
}

}

}
else
{
	if(input[i]>(545+offset3))
{
	if(input[i]==(547+offset3))
{
	vR[i] = 547;
}
else
{
	vR[i] = 546;
}

}
else
{
	if(input[i]==(545+offset3))
{
	vR[i] = 545;
}
else
{
	vR[i] = 544;
}

}

}

}

}

}
else
{
	if(input[i]>(527+offset3))
{
	if(input[i]>(535+offset3))
{
	if(input[i]>(539+offset3))
{
	if(input[i]>(541+offset3))
{
	if(input[i]==(543+offset3))
{
	vR[i] = 543;
}
else
{
	vR[i] = 542;
}

}
else
{
	if(input[i]==(541+offset3))
{
	vR[i] = 541;
}
else
{
	vR[i] = 540;
}

}

}
else
{
	if(input[i]>(537+offset3))
{
	if(input[i]==(539+offset3))
{
	vR[i] = 539;
}
else
{
	vR[i] = 538;
}

}
else
{
	if(input[i]==(537+offset3))
{
	vR[i] = 537;
}
else
{
	vR[i] = 536;
}

}

}

}
else
{
	if(input[i]>(531+offset3))
{
	if(input[i]>(533+offset3))
{
	if(input[i]==(535+offset3))
{
	vR[i] = 535;
}
else
{
	vR[i] = 534;
}

}
else
{
	if(input[i]==(533+offset3))
{
	vR[i] = 533;
}
else
{
	vR[i] = 532;
}

}

}
else
{
	if(input[i]>(529+offset3))
{
	if(input[i]==(531+offset3))
{
	vR[i] = 531;
}
else
{
	vR[i] = 530;
}

}
else
{
	if(input[i]==(529+offset3))
{
	vR[i] = 529;
}
else
{
	vR[i] = 528;
}

}

}

}

}
else
{
	if(input[i]>(519+offset3))
{
	if(input[i]>(523+offset3))
{
	if(input[i]>(525+offset3))
{
	if(input[i]==(527+offset3))
{
	vR[i] = 527;
}
else
{
	vR[i] = 526;
}

}
else
{
	if(input[i]==(525+offset3))
{
	vR[i] = 525;
}
else
{
	vR[i] = 524;
}

}

}
else
{
	if(input[i]>(521+offset3))
{
	if(input[i]==(523+offset3))
{
	vR[i] = 523;
}
else
{
	vR[i] = 522;
}

}
else
{
	if(input[i]==(521+offset3))
{
	vR[i] = 521;
}
else
{
	vR[i] = 520;
}

}

}

}
else
{
	if(input[i]>(515+offset3))
{
	if(input[i]>(517+offset3))
{
	if(input[i]==(519+offset3))
{
	vR[i] = 519;
}
else
{
	vR[i] = 518;
}

}
else
{
	if(input[i]==(517+offset3))
{
	vR[i] = 517;
}
else
{
	vR[i] = 516;
}

}

}
else
{
	if(input[i]>(513+offset3))
{
	if(input[i]==(515+offset3))
{
	vR[i] = 515;
}
else
{
	vR[i] = 514;
}

}
else
{
	if(input[i]==(513+offset3))
{
	vR[i] = 513;
}
else
{
	vR[i] = 512;
}

}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(255+offset3))
{
	if(input[i]>(383+offset3))
{
	if(input[i]>(447+offset3))
{
	if(input[i]>(479+offset3))
{
	if(input[i]>(495+offset3))
{
	if(input[i]>(503+offset3))
{
	if(input[i]>(507+offset3))
{
	if(input[i]>(509+offset3))
{
	if(input[i]==(511+offset3))
{
	vR[i] = 511;
}
else
{
	vR[i] = 510;
}

}
else
{
	if(input[i]==(509+offset3))
{
	vR[i] = 509;
}
else
{
	vR[i] = 508;
}

}

}
else
{
	if(input[i]>(505+offset3))
{
	if(input[i]==(507+offset3))
{
	vR[i] = 507;
}
else
{
	vR[i] = 506;
}

}
else
{
	if(input[i]==(505+offset3))
{
	vR[i] = 505;
}
else
{
	vR[i] = 504;
}

}

}

}
else
{
	if(input[i]>(499+offset3))
{
	if(input[i]>(501+offset3))
{
	if(input[i]==(503+offset3))
{
	vR[i] = 503;
}
else
{
	vR[i] = 502;
}

}
else
{
	if(input[i]==(501+offset3))
{
	vR[i] = 501;
}
else
{
	vR[i] = 500;
}

}

}
else
{
	if(input[i]>(497+offset3))
{
	if(input[i]==(499+offset3))
{
	vR[i] = 499;
}
else
{
	vR[i] = 498;
}

}
else
{
	if(input[i]==(497+offset3))
{
	vR[i] = 497;
}
else
{
	vR[i] = 496;
}

}

}

}

}
else
{
	if(input[i]>(487+offset3))
{
	if(input[i]>(491+offset3))
{
	if(input[i]>(493+offset3))
{
	if(input[i]==(495+offset3))
{
	vR[i] = 495;
}
else
{
	vR[i] = 494;
}

}
else
{
	if(input[i]==(493+offset3))
{
	vR[i] = 493;
}
else
{
	vR[i] = 492;
}

}

}
else
{
	if(input[i]>(489+offset3))
{
	if(input[i]==(491+offset3))
{
	vR[i] = 491;
}
else
{
	vR[i] = 490;
}

}
else
{
	if(input[i]==(489+offset3))
{
	vR[i] = 489;
}
else
{
	vR[i] = 488;
}

}

}

}
else
{
	if(input[i]>(483+offset3))
{
	if(input[i]>(485+offset3))
{
	if(input[i]==(487+offset3))
{
	vR[i] = 487;
}
else
{
	vR[i] = 486;
}

}
else
{
	if(input[i]==(485+offset3))
{
	vR[i] = 485;
}
else
{
	vR[i] = 484;
}

}

}
else
{
	if(input[i]>(481+offset3))
{
	if(input[i]==(483+offset3))
{
	vR[i] = 483;
}
else
{
	vR[i] = 482;
}

}
else
{
	if(input[i]==(481+offset3))
{
	vR[i] = 481;
}
else
{
	vR[i] = 480;
}

}

}

}

}

}
else
{
	if(input[i]>(463+offset3))
{
	if(input[i]>(471+offset3))
{
	if(input[i]>(475+offset3))
{
	if(input[i]>(477+offset3))
{
	if(input[i]==(479+offset3))
{
	vR[i] = 479;
}
else
{
	vR[i] = 478;
}

}
else
{
	if(input[i]==(477+offset3))
{
	vR[i] = 477;
}
else
{
	vR[i] = 476;
}

}

}
else
{
	if(input[i]>(473+offset3))
{
	if(input[i]==(475+offset3))
{
	vR[i] = 475;
}
else
{
	vR[i] = 474;
}

}
else
{
	if(input[i]==(473+offset3))
{
	vR[i] = 473;
}
else
{
	vR[i] = 472;
}

}

}

}
else
{
	if(input[i]>(467+offset3))
{
	if(input[i]>(469+offset3))
{
	if(input[i]==(471+offset3))
{
	vR[i] = 471;
}
else
{
	vR[i] = 470;
}

}
else
{
	if(input[i]==(469+offset3))
{
	vR[i] = 469;
}
else
{
	vR[i] = 468;
}

}

}
else
{
	if(input[i]>(465+offset3))
{
	if(input[i]==(467+offset3))
{
	vR[i] = 467;
}
else
{
	vR[i] = 466;
}

}
else
{
	if(input[i]==(465+offset3))
{
	vR[i] = 465;
}
else
{
	vR[i] = 464;
}

}

}

}

}
else
{
	if(input[i]>(455+offset3))
{
	if(input[i]>(459+offset3))
{
	if(input[i]>(461+offset3))
{
	if(input[i]==(463+offset3))
{
	vR[i] = 463;
}
else
{
	vR[i] = 462;
}

}
else
{
	if(input[i]==(461+offset3))
{
	vR[i] = 461;
}
else
{
	vR[i] = 460;
}

}

}
else
{
	if(input[i]>(457+offset3))
{
	if(input[i]==(459+offset3))
{
	vR[i] = 459;
}
else
{
	vR[i] = 458;
}

}
else
{
	if(input[i]==(457+offset3))
{
	vR[i] = 457;
}
else
{
	vR[i] = 456;
}

}

}

}
else
{
	if(input[i]>(451+offset3))
{
	if(input[i]>(453+offset3))
{
	if(input[i]==(455+offset3))
{
	vR[i] = 455;
}
else
{
	vR[i] = 454;
}

}
else
{
	if(input[i]==(453+offset3))
{
	vR[i] = 453;
}
else
{
	vR[i] = 452;
}

}

}
else
{
	if(input[i]>(449+offset3))
{
	if(input[i]==(451+offset3))
{
	vR[i] = 451;
}
else
{
	vR[i] = 450;
}

}
else
{
	if(input[i]==(449+offset3))
{
	vR[i] = 449;
}
else
{
	vR[i] = 448;
}

}

}

}

}

}

}
else
{
	if(input[i]>(415+offset3))
{
	if(input[i]>(431+offset3))
{
	if(input[i]>(439+offset3))
{
	if(input[i]>(443+offset3))
{
	if(input[i]>(445+offset3))
{
	if(input[i]==(447+offset3))
{
	vR[i] = 447;
}
else
{
	vR[i] = 446;
}

}
else
{
	if(input[i]==(445+offset3))
{
	vR[i] = 445;
}
else
{
	vR[i] = 444;
}

}

}
else
{
	if(input[i]>(441+offset3))
{
	if(input[i]==(443+offset3))
{
	vR[i] = 443;
}
else
{
	vR[i] = 442;
}

}
else
{
	if(input[i]==(441+offset3))
{
	vR[i] = 441;
}
else
{
	vR[i] = 440;
}

}

}

}
else
{
	if(input[i]>(435+offset3))
{
	if(input[i]>(437+offset3))
{
	if(input[i]==(439+offset3))
{
	vR[i] = 439;
}
else
{
	vR[i] = 438;
}

}
else
{
	if(input[i]==(437+offset3))
{
	vR[i] = 437;
}
else
{
	vR[i] = 436;
}

}

}
else
{
	if(input[i]>(433+offset3))
{
	if(input[i]==(435+offset3))
{
	vR[i] = 435;
}
else
{
	vR[i] = 434;
}

}
else
{
	if(input[i]==(433+offset3))
{
	vR[i] = 433;
}
else
{
	vR[i] = 432;
}

}

}

}

}
else
{
	if(input[i]>(423+offset3))
{
	if(input[i]>(427+offset3))
{
	if(input[i]>(429+offset3))
{
	if(input[i]==(431+offset3))
{
	vR[i] = 431;
}
else
{
	vR[i] = 430;
}

}
else
{
	if(input[i]==(429+offset3))
{
	vR[i] = 429;
}
else
{
	vR[i] = 428;
}

}

}
else
{
	if(input[i]>(425+offset3))
{
	if(input[i]==(427+offset3))
{
	vR[i] = 427;
}
else
{
	vR[i] = 426;
}

}
else
{
	if(input[i]==(425+offset3))
{
	vR[i] = 425;
}
else
{
	vR[i] = 424;
}

}

}

}
else
{
	if(input[i]>(419+offset3))
{
	if(input[i]>(421+offset3))
{
	if(input[i]==(423+offset3))
{
	vR[i] = 423;
}
else
{
	vR[i] = 422;
}

}
else
{
	if(input[i]==(421+offset3))
{
	vR[i] = 421;
}
else
{
	vR[i] = 420;
}

}

}
else
{
	if(input[i]>(417+offset3))
{
	if(input[i]==(419+offset3))
{
	vR[i] = 419;
}
else
{
	vR[i] = 418;
}

}
else
{
	if(input[i]==(417+offset3))
{
	vR[i] = 417;
}
else
{
	vR[i] = 416;
}

}

}

}

}

}
else
{
	if(input[i]>(399+offset3))
{
	if(input[i]>(407+offset3))
{
	if(input[i]>(411+offset3))
{
	if(input[i]>(413+offset3))
{
	if(input[i]==(415+offset3))
{
	vR[i] = 415;
}
else
{
	vR[i] = 414;
}

}
else
{
	if(input[i]==(413+offset3))
{
	vR[i] = 413;
}
else
{
	vR[i] = 412;
}

}

}
else
{
	if(input[i]>(409+offset3))
{
	if(input[i]==(411+offset3))
{
	vR[i] = 411;
}
else
{
	vR[i] = 410;
}

}
else
{
	if(input[i]==(409+offset3))
{
	vR[i] = 409;
}
else
{
	vR[i] = 408;
}

}

}

}
else
{
	if(input[i]>(403+offset3))
{
	if(input[i]>(405+offset3))
{
	if(input[i]==(407+offset3))
{
	vR[i] = 407;
}
else
{
	vR[i] = 406;
}

}
else
{
	if(input[i]==(405+offset3))
{
	vR[i] = 405;
}
else
{
	vR[i] = 404;
}

}

}
else
{
	if(input[i]>(401+offset3))
{
	if(input[i]==(403+offset3))
{
	vR[i] = 403;
}
else
{
	vR[i] = 402;
}

}
else
{
	if(input[i]==(401+offset3))
{
	vR[i] = 401;
}
else
{
	vR[i] = 400;
}

}

}

}

}
else
{
	if(input[i]>(391+offset3))
{
	if(input[i]>(395+offset3))
{
	if(input[i]>(397+offset3))
{
	if(input[i]==(399+offset3))
{
	vR[i] = 399;
}
else
{
	vR[i] = 398;
}

}
else
{
	if(input[i]==(397+offset3))
{
	vR[i] = 397;
}
else
{
	vR[i] = 396;
}

}

}
else
{
	if(input[i]>(393+offset3))
{
	if(input[i]==(395+offset3))
{
	vR[i] = 395;
}
else
{
	vR[i] = 394;
}

}
else
{
	if(input[i]==(393+offset3))
{
	vR[i] = 393;
}
else
{
	vR[i] = 392;
}

}

}

}
else
{
	if(input[i]>(387+offset3))
{
	if(input[i]>(389+offset3))
{
	if(input[i]==(391+offset3))
{
	vR[i] = 391;
}
else
{
	vR[i] = 390;
}

}
else
{
	if(input[i]==(389+offset3))
{
	vR[i] = 389;
}
else
{
	vR[i] = 388;
}

}

}
else
{
	if(input[i]>(385+offset3))
{
	if(input[i]==(387+offset3))
{
	vR[i] = 387;
}
else
{
	vR[i] = 386;
}

}
else
{
	if(input[i]==(385+offset3))
{
	vR[i] = 385;
}
else
{
	vR[i] = 384;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(319+offset3))
{
	if(input[i]>(351+offset3))
{
	if(input[i]>(367+offset3))
{
	if(input[i]>(375+offset3))
{
	if(input[i]>(379+offset3))
{
	if(input[i]>(381+offset3))
{
	if(input[i]==(383+offset3))
{
	vR[i] = 383;
}
else
{
	vR[i] = 382;
}

}
else
{
	if(input[i]==(381+offset3))
{
	vR[i] = 381;
}
else
{
	vR[i] = 380;
}

}

}
else
{
	if(input[i]>(377+offset3))
{
	if(input[i]==(379+offset3))
{
	vR[i] = 379;
}
else
{
	vR[i] = 378;
}

}
else
{
	if(input[i]==(377+offset3))
{
	vR[i] = 377;
}
else
{
	vR[i] = 376;
}

}

}

}
else
{
	if(input[i]>(371+offset3))
{
	if(input[i]>(373+offset3))
{
	if(input[i]==(375+offset3))
{
	vR[i] = 375;
}
else
{
	vR[i] = 374;
}

}
else
{
	if(input[i]==(373+offset3))
{
	vR[i] = 373;
}
else
{
	vR[i] = 372;
}

}

}
else
{
	if(input[i]>(369+offset3))
{
	if(input[i]==(371+offset3))
{
	vR[i] = 371;
}
else
{
	vR[i] = 370;
}

}
else
{
	if(input[i]==(369+offset3))
{
	vR[i] = 369;
}
else
{
	vR[i] = 368;
}

}

}

}

}
else
{
	if(input[i]>(359+offset3))
{
	if(input[i]>(363+offset3))
{
	if(input[i]>(365+offset3))
{
	if(input[i]==(367+offset3))
{
	vR[i] = 367;
}
else
{
	vR[i] = 366;
}

}
else
{
	if(input[i]==(365+offset3))
{
	vR[i] = 365;
}
else
{
	vR[i] = 364;
}

}

}
else
{
	if(input[i]>(361+offset3))
{
	if(input[i]==(363+offset3))
{
	vR[i] = 363;
}
else
{
	vR[i] = 362;
}

}
else
{
	if(input[i]==(361+offset3))
{
	vR[i] = 361;
}
else
{
	vR[i] = 360;
}

}

}

}
else
{
	if(input[i]>(355+offset3))
{
	if(input[i]>(357+offset3))
{
	if(input[i]==(359+offset3))
{
	vR[i] = 359;
}
else
{
	vR[i] = 358;
}

}
else
{
	if(input[i]==(357+offset3))
{
	vR[i] = 357;
}
else
{
	vR[i] = 356;
}

}

}
else
{
	if(input[i]>(353+offset3))
{
	if(input[i]==(355+offset3))
{
	vR[i] = 355;
}
else
{
	vR[i] = 354;
}

}
else
{
	if(input[i]==(353+offset3))
{
	vR[i] = 353;
}
else
{
	vR[i] = 352;
}

}

}

}

}

}
else
{
	if(input[i]>(335+offset3))
{
	if(input[i]>(343+offset3))
{
	if(input[i]>(347+offset3))
{
	if(input[i]>(349+offset3))
{
	if(input[i]==(351+offset3))
{
	vR[i] = 351;
}
else
{
	vR[i] = 350;
}

}
else
{
	if(input[i]==(349+offset3))
{
	vR[i] = 349;
}
else
{
	vR[i] = 348;
}

}

}
else
{
	if(input[i]>(345+offset3))
{
	if(input[i]==(347+offset3))
{
	vR[i] = 347;
}
else
{
	vR[i] = 346;
}

}
else
{
	if(input[i]==(345+offset3))
{
	vR[i] = 345;
}
else
{
	vR[i] = 344;
}

}

}

}
else
{
	if(input[i]>(339+offset3))
{
	if(input[i]>(341+offset3))
{
	if(input[i]==(343+offset3))
{
	vR[i] = 343;
}
else
{
	vR[i] = 342;
}

}
else
{
	if(input[i]==(341+offset3))
{
	vR[i] = 341;
}
else
{
	vR[i] = 340;
}

}

}
else
{
	if(input[i]>(337+offset3))
{
	if(input[i]==(339+offset3))
{
	vR[i] = 339;
}
else
{
	vR[i] = 338;
}

}
else
{
	if(input[i]==(337+offset3))
{
	vR[i] = 337;
}
else
{
	vR[i] = 336;
}

}

}

}

}
else
{
	if(input[i]>(327+offset3))
{
	if(input[i]>(331+offset3))
{
	if(input[i]>(333+offset3))
{
	if(input[i]==(335+offset3))
{
	vR[i] = 335;
}
else
{
	vR[i] = 334;
}

}
else
{
	if(input[i]==(333+offset3))
{
	vR[i] = 333;
}
else
{
	vR[i] = 332;
}

}

}
else
{
	if(input[i]>(329+offset3))
{
	if(input[i]==(331+offset3))
{
	vR[i] = 331;
}
else
{
	vR[i] = 330;
}

}
else
{
	if(input[i]==(329+offset3))
{
	vR[i] = 329;
}
else
{
	vR[i] = 328;
}

}

}

}
else
{
	if(input[i]>(323+offset3))
{
	if(input[i]>(325+offset3))
{
	if(input[i]==(327+offset3))
{
	vR[i] = 327;
}
else
{
	vR[i] = 326;
}

}
else
{
	if(input[i]==(325+offset3))
{
	vR[i] = 325;
}
else
{
	vR[i] = 324;
}

}

}
else
{
	if(input[i]>(321+offset3))
{
	if(input[i]==(323+offset3))
{
	vR[i] = 323;
}
else
{
	vR[i] = 322;
}

}
else
{
	if(input[i]==(321+offset3))
{
	vR[i] = 321;
}
else
{
	vR[i] = 320;
}

}

}

}

}

}

}
else
{
	if(input[i]>(287+offset3))
{
	if(input[i]>(303+offset3))
{
	if(input[i]>(311+offset3))
{
	if(input[i]>(315+offset3))
{
	if(input[i]>(317+offset3))
{
	if(input[i]==(319+offset3))
{
	vR[i] = 319;
}
else
{
	vR[i] = 318;
}

}
else
{
	if(input[i]==(317+offset3))
{
	vR[i] = 317;
}
else
{
	vR[i] = 316;
}

}

}
else
{
	if(input[i]>(313+offset3))
{
	if(input[i]==(315+offset3))
{
	vR[i] = 315;
}
else
{
	vR[i] = 314;
}

}
else
{
	if(input[i]==(313+offset3))
{
	vR[i] = 313;
}
else
{
	vR[i] = 312;
}

}

}

}
else
{
	if(input[i]>(307+offset3))
{
	if(input[i]>(309+offset3))
{
	if(input[i]==(311+offset3))
{
	vR[i] = 311;
}
else
{
	vR[i] = 310;
}

}
else
{
	if(input[i]==(309+offset3))
{
	vR[i] = 309;
}
else
{
	vR[i] = 308;
}

}

}
else
{
	if(input[i]>(305+offset3))
{
	if(input[i]==(307+offset3))
{
	vR[i] = 307;
}
else
{
	vR[i] = 306;
}

}
else
{
	if(input[i]==(305+offset3))
{
	vR[i] = 305;
}
else
{
	vR[i] = 304;
}

}

}

}

}
else
{
	if(input[i]>(295+offset3))
{
	if(input[i]>(299+offset3))
{
	if(input[i]>(301+offset3))
{
	if(input[i]==(303+offset3))
{
	vR[i] = 303;
}
else
{
	vR[i] = 302;
}

}
else
{
	if(input[i]==(301+offset3))
{
	vR[i] = 301;
}
else
{
	vR[i] = 300;
}

}

}
else
{
	if(input[i]>(297+offset3))
{
	if(input[i]==(299+offset3))
{
	vR[i] = 299;
}
else
{
	vR[i] = 298;
}

}
else
{
	if(input[i]==(297+offset3))
{
	vR[i] = 297;
}
else
{
	vR[i] = 296;
}

}

}

}
else
{
	if(input[i]>(291+offset3))
{
	if(input[i]>(293+offset3))
{
	if(input[i]==(295+offset3))
{
	vR[i] = 295;
}
else
{
	vR[i] = 294;
}

}
else
{
	if(input[i]==(293+offset3))
{
	vR[i] = 293;
}
else
{
	vR[i] = 292;
}

}

}
else
{
	if(input[i]>(289+offset3))
{
	if(input[i]==(291+offset3))
{
	vR[i] = 291;
}
else
{
	vR[i] = 290;
}

}
else
{
	if(input[i]==(289+offset3))
{
	vR[i] = 289;
}
else
{
	vR[i] = 288;
}

}

}

}

}

}
else
{
	if(input[i]>(271+offset3))
{
	if(input[i]>(279+offset3))
{
	if(input[i]>(283+offset3))
{
	if(input[i]>(285+offset3))
{
	if(input[i]==(287+offset3))
{
	vR[i] = 287;
}
else
{
	vR[i] = 286;
}

}
else
{
	if(input[i]==(285+offset3))
{
	vR[i] = 285;
}
else
{
	vR[i] = 284;
}

}

}
else
{
	if(input[i]>(281+offset3))
{
	if(input[i]==(283+offset3))
{
	vR[i] = 283;
}
else
{
	vR[i] = 282;
}

}
else
{
	if(input[i]==(281+offset3))
{
	vR[i] = 281;
}
else
{
	vR[i] = 280;
}

}

}

}
else
{
	if(input[i]>(275+offset3))
{
	if(input[i]>(277+offset3))
{
	if(input[i]==(279+offset3))
{
	vR[i] = 279;
}
else
{
	vR[i] = 278;
}

}
else
{
	if(input[i]==(277+offset3))
{
	vR[i] = 277;
}
else
{
	vR[i] = 276;
}

}

}
else
{
	if(input[i]>(273+offset3))
{
	if(input[i]==(275+offset3))
{
	vR[i] = 275;
}
else
{
	vR[i] = 274;
}

}
else
{
	if(input[i]==(273+offset3))
{
	vR[i] = 273;
}
else
{
	vR[i] = 272;
}

}

}

}

}
else
{
	if(input[i]>(263+offset3))
{
	if(input[i]>(267+offset3))
{
	if(input[i]>(269+offset3))
{
	if(input[i]==(271+offset3))
{
	vR[i] = 271;
}
else
{
	vR[i] = 270;
}

}
else
{
	if(input[i]==(269+offset3))
{
	vR[i] = 269;
}
else
{
	vR[i] = 268;
}

}

}
else
{
	if(input[i]>(265+offset3))
{
	if(input[i]==(267+offset3))
{
	vR[i] = 267;
}
else
{
	vR[i] = 266;
}

}
else
{
	if(input[i]==(265+offset3))
{
	vR[i] = 265;
}
else
{
	vR[i] = 264;
}

}

}

}
else
{
	if(input[i]>(259+offset3))
{
	if(input[i]>(261+offset3))
{
	if(input[i]==(263+offset3))
{
	vR[i] = 263;
}
else
{
	vR[i] = 262;
}

}
else
{
	if(input[i]==(261+offset3))
{
	vR[i] = 261;
}
else
{
	vR[i] = 260;
}

}

}
else
{
	if(input[i]>(257+offset3))
{
	if(input[i]==(259+offset3))
{
	vR[i] = 259;
}
else
{
	vR[i] = 258;
}

}
else
{
	if(input[i]==(257+offset3))
{
	vR[i] = 257;
}
else
{
	vR[i] = 256;
}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(127+offset3))
{
	if(input[i]>(191+offset3))
{
	if(input[i]>(223+offset3))
{
	if(input[i]>(239+offset3))
{
	if(input[i]>(247+offset3))
{
	if(input[i]>(251+offset3))
{
	if(input[i]>(253+offset3))
{
	if(input[i]==(255+offset3))
{
	vR[i] = 255;
}
else
{
	vR[i] = 254;
}

}
else
{
	if(input[i]==(253+offset3))
{
	vR[i] = 253;
}
else
{
	vR[i] = 252;
}

}

}
else
{
	if(input[i]>(249+offset3))
{
	if(input[i]==(251+offset3))
{
	vR[i] = 251;
}
else
{
	vR[i] = 250;
}

}
else
{
	if(input[i]==(249+offset3))
{
	vR[i] = 249;
}
else
{
	vR[i] = 248;
}

}

}

}
else
{
	if(input[i]>(243+offset3))
{
	if(input[i]>(245+offset3))
{
	if(input[i]==(247+offset3))
{
	vR[i] = 247;
}
else
{
	vR[i] = 246;
}

}
else
{
	if(input[i]==(245+offset3))
{
	vR[i] = 245;
}
else
{
	vR[i] = 244;
}

}

}
else
{
	if(input[i]>(241+offset3))
{
	if(input[i]==(243+offset3))
{
	vR[i] = 243;
}
else
{
	vR[i] = 242;
}

}
else
{
	if(input[i]==(241+offset3))
{
	vR[i] = 241;
}
else
{
	vR[i] = 240;
}

}

}

}

}
else
{
	if(input[i]>(231+offset3))
{
	if(input[i]>(235+offset3))
{
	if(input[i]>(237+offset3))
{
	if(input[i]==(239+offset3))
{
	vR[i] = 239;
}
else
{
	vR[i] = 238;
}

}
else
{
	if(input[i]==(237+offset3))
{
	vR[i] = 237;
}
else
{
	vR[i] = 236;
}

}

}
else
{
	if(input[i]>(233+offset3))
{
	if(input[i]==(235+offset3))
{
	vR[i] = 235;
}
else
{
	vR[i] = 234;
}

}
else
{
	if(input[i]==(233+offset3))
{
	vR[i] = 233;
}
else
{
	vR[i] = 232;
}

}

}

}
else
{
	if(input[i]>(227+offset3))
{
	if(input[i]>(229+offset3))
{
	if(input[i]==(231+offset3))
{
	vR[i] = 231;
}
else
{
	vR[i] = 230;
}

}
else
{
	if(input[i]==(229+offset3))
{
	vR[i] = 229;
}
else
{
	vR[i] = 228;
}

}

}
else
{
	if(input[i]>(225+offset3))
{
	if(input[i]==(227+offset3))
{
	vR[i] = 227;
}
else
{
	vR[i] = 226;
}

}
else
{
	if(input[i]==(225+offset3))
{
	vR[i] = 225;
}
else
{
	vR[i] = 224;
}

}

}

}

}

}
else
{
	if(input[i]>(207+offset3))
{
	if(input[i]>(215+offset3))
{
	if(input[i]>(219+offset3))
{
	if(input[i]>(221+offset3))
{
	if(input[i]==(223+offset3))
{
	vR[i] = 223;
}
else
{
	vR[i] = 222;
}

}
else
{
	if(input[i]==(221+offset3))
{
	vR[i] = 221;
}
else
{
	vR[i] = 220;
}

}

}
else
{
	if(input[i]>(217+offset3))
{
	if(input[i]==(219+offset3))
{
	vR[i] = 219;
}
else
{
	vR[i] = 218;
}

}
else
{
	if(input[i]==(217+offset3))
{
	vR[i] = 217;
}
else
{
	vR[i] = 216;
}

}

}

}
else
{
	if(input[i]>(211+offset3))
{
	if(input[i]>(213+offset3))
{
	if(input[i]==(215+offset3))
{
	vR[i] = 215;
}
else
{
	vR[i] = 214;
}

}
else
{
	if(input[i]==(213+offset3))
{
	vR[i] = 213;
}
else
{
	vR[i] = 212;
}

}

}
else
{
	if(input[i]>(209+offset3))
{
	if(input[i]==(211+offset3))
{
	vR[i] = 211;
}
else
{
	vR[i] = 210;
}

}
else
{
	if(input[i]==(209+offset3))
{
	vR[i] = 209;
}
else
{
	vR[i] = 208;
}

}

}

}

}
else
{
	if(input[i]>(199+offset3))
{
	if(input[i]>(203+offset3))
{
	if(input[i]>(205+offset3))
{
	if(input[i]==(207+offset3))
{
	vR[i] = 207;
}
else
{
	vR[i] = 206;
}

}
else
{
	if(input[i]==(205+offset3))
{
	vR[i] = 205;
}
else
{
	vR[i] = 204;
}

}

}
else
{
	if(input[i]>(201+offset3))
{
	if(input[i]==(203+offset3))
{
	vR[i] = 203;
}
else
{
	vR[i] = 202;
}

}
else
{
	if(input[i]==(201+offset3))
{
	vR[i] = 201;
}
else
{
	vR[i] = 200;
}

}

}

}
else
{
	if(input[i]>(195+offset3))
{
	if(input[i]>(197+offset3))
{
	if(input[i]==(199+offset3))
{
	vR[i] = 199;
}
else
{
	vR[i] = 198;
}

}
else
{
	if(input[i]==(197+offset3))
{
	vR[i] = 197;
}
else
{
	vR[i] = 196;
}

}

}
else
{
	if(input[i]>(193+offset3))
{
	if(input[i]==(195+offset3))
{
	vR[i] = 195;
}
else
{
	vR[i] = 194;
}

}
else
{
	if(input[i]==(193+offset3))
{
	vR[i] = 193;
}
else
{
	vR[i] = 192;
}

}

}

}

}

}

}
else
{
	if(input[i]>(159+offset3))
{
	if(input[i]>(175+offset3))
{
	if(input[i]>(183+offset3))
{
	if(input[i]>(187+offset3))
{
	if(input[i]>(189+offset3))
{
	if(input[i]==(191+offset3))
{
	vR[i] = 191;
}
else
{
	vR[i] = 190;
}

}
else
{
	if(input[i]==(189+offset3))
{
	vR[i] = 189;
}
else
{
	vR[i] = 188;
}

}

}
else
{
	if(input[i]>(185+offset3))
{
	if(input[i]==(187+offset3))
{
	vR[i] = 187;
}
else
{
	vR[i] = 186;
}

}
else
{
	if(input[i]==(185+offset3))
{
	vR[i] = 185;
}
else
{
	vR[i] = 184;
}

}

}

}
else
{
	if(input[i]>(179+offset3))
{
	if(input[i]>(181+offset3))
{
	if(input[i]==(183+offset3))
{
	vR[i] = 183;
}
else
{
	vR[i] = 182;
}

}
else
{
	if(input[i]==(181+offset3))
{
	vR[i] = 181;
}
else
{
	vR[i] = 180;
}

}

}
else
{
	if(input[i]>(177+offset3))
{
	if(input[i]==(179+offset3))
{
	vR[i] = 179;
}
else
{
	vR[i] = 178;
}

}
else
{
	if(input[i]==(177+offset3))
{
	vR[i] = 177;
}
else
{
	vR[i] = 176;
}

}

}

}

}
else
{
	if(input[i]>(167+offset3))
{
	if(input[i]>(171+offset3))
{
	if(input[i]>(173+offset3))
{
	if(input[i]==(175+offset3))
{
	vR[i] = 175;
}
else
{
	vR[i] = 174;
}

}
else
{
	if(input[i]==(173+offset3))
{
	vR[i] = 173;
}
else
{
	vR[i] = 172;
}

}

}
else
{
	if(input[i]>(169+offset3))
{
	if(input[i]==(171+offset3))
{
	vR[i] = 171;
}
else
{
	vR[i] = 170;
}

}
else
{
	if(input[i]==(169+offset3))
{
	vR[i] = 169;
}
else
{
	vR[i] = 168;
}

}

}

}
else
{
	if(input[i]>(163+offset3))
{
	if(input[i]>(165+offset3))
{
	if(input[i]==(167+offset3))
{
	vR[i] = 167;
}
else
{
	vR[i] = 166;
}

}
else
{
	if(input[i]==(165+offset3))
{
	vR[i] = 165;
}
else
{
	vR[i] = 164;
}

}

}
else
{
	if(input[i]>(161+offset3))
{
	if(input[i]==(163+offset3))
{
	vR[i] = 163;
}
else
{
	vR[i] = 162;
}

}
else
{
	if(input[i]==(161+offset3))
{
	vR[i] = 161;
}
else
{
	vR[i] = 160;
}

}

}

}

}

}
else
{
	if(input[i]>(143+offset3))
{
	if(input[i]>(151+offset3))
{
	if(input[i]>(155+offset3))
{
	if(input[i]>(157+offset3))
{
	if(input[i]==(159+offset3))
{
	vR[i] = 159;
}
else
{
	vR[i] = 158;
}

}
else
{
	if(input[i]==(157+offset3))
{
	vR[i] = 157;
}
else
{
	vR[i] = 156;
}

}

}
else
{
	if(input[i]>(153+offset3))
{
	if(input[i]==(155+offset3))
{
	vR[i] = 155;
}
else
{
	vR[i] = 154;
}

}
else
{
	if(input[i]==(153+offset3))
{
	vR[i] = 153;
}
else
{
	vR[i] = 152;
}

}

}

}
else
{
	if(input[i]>(147+offset3))
{
	if(input[i]>(149+offset3))
{
	if(input[i]==(151+offset3))
{
	vR[i] = 151;
}
else
{
	vR[i] = 150;
}

}
else
{
	if(input[i]==(149+offset3))
{
	vR[i] = 149;
}
else
{
	vR[i] = 148;
}

}

}
else
{
	if(input[i]>(145+offset3))
{
	if(input[i]==(147+offset3))
{
	vR[i] = 147;
}
else
{
	vR[i] = 146;
}

}
else
{
	if(input[i]==(145+offset3))
{
	vR[i] = 145;
}
else
{
	vR[i] = 144;
}

}

}

}

}
else
{
	if(input[i]>(135+offset3))
{
	if(input[i]>(139+offset3))
{
	if(input[i]>(141+offset3))
{
	if(input[i]==(143+offset3))
{
	vR[i] = 143;
}
else
{
	vR[i] = 142;
}

}
else
{
	if(input[i]==(141+offset3))
{
	vR[i] = 141;
}
else
{
	vR[i] = 140;
}

}

}
else
{
	if(input[i]>(137+offset3))
{
	if(input[i]==(139+offset3))
{
	vR[i] = 139;
}
else
{
	vR[i] = 138;
}

}
else
{
	if(input[i]==(137+offset3))
{
	vR[i] = 137;
}
else
{
	vR[i] = 136;
}

}

}

}
else
{
	if(input[i]>(131+offset3))
{
	if(input[i]>(133+offset3))
{
	if(input[i]==(135+offset3))
{
	vR[i] = 135;
}
else
{
	vR[i] = 134;
}

}
else
{
	if(input[i]==(133+offset3))
{
	vR[i] = 133;
}
else
{
	vR[i] = 132;
}

}

}
else
{
	if(input[i]>(129+offset3))
{
	if(input[i]==(131+offset3))
{
	vR[i] = 131;
}
else
{
	vR[i] = 130;
}

}
else
{
	if(input[i]==(129+offset3))
{
	vR[i] = 129;
}
else
{
	vR[i] = 128;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(63+offset3))
{
	if(input[i]>(95+offset3))
{
	if(input[i]>(111+offset3))
{
	if(input[i]>(119+offset3))
{
	if(input[i]>(123+offset3))
{
	if(input[i]>(125+offset3))
{
	if(input[i]==(127+offset3))
{
	vR[i] = 127;
}
else
{
	vR[i] = 126;
}

}
else
{
	if(input[i]==(125+offset3))
{
	vR[i] = 125;
}
else
{
	vR[i] = 124;
}

}

}
else
{
	if(input[i]>(121+offset3))
{
	if(input[i]==(123+offset3))
{
	vR[i] = 123;
}
else
{
	vR[i] = 122;
}

}
else
{
	if(input[i]==(121+offset3))
{
	vR[i] = 121;
}
else
{
	vR[i] = 120;
}

}

}

}
else
{
	if(input[i]>(115+offset3))
{
	if(input[i]>(117+offset3))
{
	if(input[i]==(119+offset3))
{
	vR[i] = 119;
}
else
{
	vR[i] = 118;
}

}
else
{
	if(input[i]==(117+offset3))
{
	vR[i] = 117;
}
else
{
	vR[i] = 116;
}

}

}
else
{
	if(input[i]>(113+offset3))
{
	if(input[i]==(115+offset3))
{
	vR[i] = 115;
}
else
{
	vR[i] = 114;
}

}
else
{
	if(input[i]==(113+offset3))
{
	vR[i] = 113;
}
else
{
	vR[i] = 112;
}

}

}

}

}
else
{
	if(input[i]>(103+offset3))
{
	if(input[i]>(107+offset3))
{
	if(input[i]>(109+offset3))
{
	if(input[i]==(111+offset3))
{
	vR[i] = 111;
}
else
{
	vR[i] = 110;
}

}
else
{
	if(input[i]==(109+offset3))
{
	vR[i] = 109;
}
else
{
	vR[i] = 108;
}

}

}
else
{
	if(input[i]>(105+offset3))
{
	if(input[i]==(107+offset3))
{
	vR[i] = 107;
}
else
{
	vR[i] = 106;
}

}
else
{
	if(input[i]==(105+offset3))
{
	vR[i] = 105;
}
else
{
	vR[i] = 104;
}

}

}

}
else
{
	if(input[i]>(99+offset3))
{
	if(input[i]>(101+offset3))
{
	if(input[i]==(103+offset3))
{
	vR[i] = 103;
}
else
{
	vR[i] = 102;
}

}
else
{
	if(input[i]==(101+offset3))
{
	vR[i] = 101;
}
else
{
	vR[i] = 100;
}

}

}
else
{
	if(input[i]>(97+offset3))
{
	if(input[i]==(99+offset3))
{
	vR[i] = 99;
}
else
{
	vR[i] = 98;
}

}
else
{
	if(input[i]==(97+offset3))
{
	vR[i] = 97;
}
else
{
	vR[i] = 96;
}

}

}

}

}

}
else
{
	if(input[i]>(79+offset3))
{
	if(input[i]>(87+offset3))
{
	if(input[i]>(91+offset3))
{
	if(input[i]>(93+offset3))
{
	if(input[i]==(95+offset3))
{
	vR[i] = 95;
}
else
{
	vR[i] = 94;
}

}
else
{
	if(input[i]==(93+offset3))
{
	vR[i] = 93;
}
else
{
	vR[i] = 92;
}

}

}
else
{
	if(input[i]>(89+offset3))
{
	if(input[i]==(91+offset3))
{
	vR[i] = 91;
}
else
{
	vR[i] = 90;
}

}
else
{
	if(input[i]==(89+offset3))
{
	vR[i] = 89;
}
else
{
	vR[i] = 88;
}

}

}

}
else
{
	if(input[i]>(83+offset3))
{
	if(input[i]>(85+offset3))
{
	if(input[i]==(87+offset3))
{
	vR[i] = 87;
}
else
{
	vR[i] = 86;
}

}
else
{
	if(input[i]==(85+offset3))
{
	vR[i] = 85;
}
else
{
	vR[i] = 84;
}

}

}
else
{
	if(input[i]>(81+offset3))
{
	if(input[i]==(83+offset3))
{
	vR[i] = 83;
}
else
{
	vR[i] = 82;
}

}
else
{
	if(input[i]==(81+offset3))
{
	vR[i] = 81;
}
else
{
	vR[i] = 80;
}

}

}

}

}
else
{
	if(input[i]>(71+offset3))
{
	if(input[i]>(75+offset3))
{
	if(input[i]>(77+offset3))
{
	if(input[i]==(79+offset3))
{
	vR[i] = 79;
}
else
{
	vR[i] = 78;
}

}
else
{
	if(input[i]==(77+offset3))
{
	vR[i] = 77;
}
else
{
	vR[i] = 76;
}

}

}
else
{
	if(input[i]>(73+offset3))
{
	if(input[i]==(75+offset3))
{
	vR[i] = 75;
}
else
{
	vR[i] = 74;
}

}
else
{
	if(input[i]==(73+offset3))
{
	vR[i] = 73;
}
else
{
	vR[i] = 72;
}

}

}

}
else
{
	if(input[i]>(67+offset3))
{
	if(input[i]>(69+offset3))
{
	if(input[i]==(71+offset3))
{
	vR[i] = 71;
}
else
{
	vR[i] = 70;
}

}
else
{
	if(input[i]==(69+offset3))
{
	vR[i] = 69;
}
else
{
	vR[i] = 68;
}

}

}
else
{
	if(input[i]>(65+offset3))
{
	if(input[i]==(67+offset3))
{
	vR[i] = 67;
}
else
{
	vR[i] = 66;
}

}
else
{
	if(input[i]==(65+offset3))
{
	vR[i] = 65;
}
else
{
	vR[i] = 64;
}

}

}

}

}

}

}
else
{
	if(input[i]>(31+offset3))
{
	if(input[i]>(47+offset3))
{
	if(input[i]>(55+offset3))
{
	if(input[i]>(59+offset3))
{
	if(input[i]>(61+offset3))
{
	if(input[i]==(63+offset3))
{
	vR[i] = 63;
}
else
{
	vR[i] = 62;
}

}
else
{
	if(input[i]==(61+offset3))
{
	vR[i] = 61;
}
else
{
	vR[i] = 60;
}

}

}
else
{
	if(input[i]>(57+offset3))
{
	if(input[i]==(59+offset3))
{
	vR[i] = 59;
}
else
{
	vR[i] = 58;
}

}
else
{
	if(input[i]==(57+offset3))
{
	vR[i] = 57;
}
else
{
	vR[i] = 56;
}

}

}

}
else
{
	if(input[i]>(51+offset3))
{
	if(input[i]>(53+offset3))
{
	if(input[i]==(55+offset3))
{
	vR[i] = 55;
}
else
{
	vR[i] = 54;
}

}
else
{
	if(input[i]==(53+offset3))
{
	vR[i] = 53;
}
else
{
	vR[i] = 52;
}

}

}
else
{
	if(input[i]>(49+offset3))
{
	if(input[i]==(51+offset3))
{
	vR[i] = 51;
}
else
{
	vR[i] = 50;
}

}
else
{
	if(input[i]==(49+offset3))
{
	vR[i] = 49;
}
else
{
	vR[i] = 48;
}

}

}

}

}
else
{
	if(input[i]>(39+offset3))
{
	if(input[i]>(43+offset3))
{
	if(input[i]>(45+offset3))
{
	if(input[i]==(47+offset3))
{
	vR[i] = 47;
}
else
{
	vR[i] = 46;
}

}
else
{
	if(input[i]==(45+offset3))
{
	vR[i] = 45;
}
else
{
	vR[i] = 44;
}

}

}
else
{
	if(input[i]>(41+offset3))
{
	if(input[i]==(43+offset3))
{
	vR[i] = 43;
}
else
{
	vR[i] = 42;
}

}
else
{
	if(input[i]==(41+offset3))
{
	vR[i] = 41;
}
else
{
	vR[i] = 40;
}

}

}

}
else
{
	if(input[i]>(35+offset3))
{
	if(input[i]>(37+offset3))
{
	if(input[i]==(39+offset3))
{
	vR[i] = 39;
}
else
{
	vR[i] = 38;
}

}
else
{
	if(input[i]==(37+offset3))
{
	vR[i] = 37;
}
else
{
	vR[i] = 36;
}

}

}
else
{
	if(input[i]>(33+offset3))
{
	if(input[i]==(35+offset3))
{
	vR[i] = 35;
}
else
{
	vR[i] = 34;
}

}
else
{
	if(input[i]==(33+offset3))
{
	vR[i] = 33;
}
else
{
	vR[i] = 32;
}

}

}

}

}

}
else
{
	if(input[i]>(15+offset3))
{
	if(input[i]>(23+offset3))
{
	if(input[i]>(27+offset3))
{
	if(input[i]>(29+offset3))
{
	if(input[i]==(31+offset3))
{
	vR[i] = 31;
}
else
{
	vR[i] = 30;
}

}
else
{
	if(input[i]==(29+offset3))
{
	vR[i] = 29;
}
else
{
	vR[i] = 28;
}

}

}
else
{
	if(input[i]>(25+offset3))
{
	if(input[i]==(27+offset3))
{
	vR[i] = 27;
}
else
{
	vR[i] = 26;
}

}
else
{
	if(input[i]==(25+offset3))
{
	vR[i] = 25;
}
else
{
	vR[i] = 24;
}

}

}

}
else
{
	if(input[i]>(19+offset3))
{
	if(input[i]>(21+offset3))
{
	if(input[i]==(23+offset3))
{
	vR[i] = 23;
}
else
{
	vR[i] = 22;
}

}
else
{
	if(input[i]==(21+offset3))
{
	vR[i] = 21;
}
else
{
	vR[i] = 20;
}

}

}
else
{
	if(input[i]>(17+offset3))
{
	if(input[i]==(19+offset3))
{
	vR[i] = 19;
}
else
{
	vR[i] = 18;
}

}
else
{
	if(input[i]==(17+offset3))
{
	vR[i] = 17;
}
else
{
	vR[i] = 16;
}

}

}

}

}
else
{
	if(input[i]>(7+offset3))
{
	if(input[i]>(11+offset3))
{
	if(input[i]>(13+offset3))
{
	if(input[i]==(15+offset3))
{
	vR[i] = 15;
}
else
{
	vR[i] = 14;
}

}
else
{
	if(input[i]==(13+offset3))
{
	vR[i] = 13;
}
else
{
	vR[i] = 12;
}

}

}
else
{
	if(input[i]>(9+offset3))
{
	if(input[i]==(11+offset3))
{
	vR[i] = 11;
}
else
{
	vR[i] = 10;
}

}
else
{
	if(input[i]==(9+offset3))
{
	vR[i] = 9;
}
else
{
	vR[i] = 8;
}

}

}

}
else
{
	if(input[i]>(3+offset3))
{
	if(input[i]>(5+offset3))
{
	if(input[i]==(7+offset3))
{
	vR[i] = 7;
}
else
{
	vR[i] = 6;
}

}
else
{
	if(input[i]==(5+offset3))
{
	vR[i] = 5;
}
else
{
	vR[i] = 4;
}

}

}
else
{
	if(input[i]>(1+offset3))
{
	if(input[i]==(3+offset3))
{
	vR[i] = 3;
}
else
{
	vR[i] = 2;
}

}
else
{
	if(input[i]==(1+offset3))
{
	vR[i] = 1;
}
else
{
	vR[i] = 0;
}

}

}

}

}

}

}

}

}

}

}


	}
}

int main(int argc, char **argv)
{
	float elapsed_time[TIMES], avg = 0.0, sd = 0.0;
	// set up device
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("[CUDA - Random Forest]: EXP %s\n", EXP_NAME);
	printf("[CUDA - Random Forest]: Using Device %d: %s\n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));

	// set up data size of vectors
	int nElem = N_ELEM;
	printf("[CUDA - Random Forest]: Vector Size %d\n", nElem);

	// malloc host memory
	size_t nBytes = nElem * sizeof(float);
	float *h_vR;
	h_vR = (float *)malloc(nBytes);
	float *h_input;
	h_input = (float *)malloc(nBytes);

	printf("[CUDA - Random Forest]: Start Reading Dataset\n");
	initData(h_input, nElem);
	printf("[CUDA - Random Forest]: Finished Reading Dataset\n");
	memset(h_vR, 0, nBytes);

	// malloc device global memory
	float *d_vR;
	CHECK(hipMalloc((float **)&d_vR, nBytes));
	float *d_input;
	CHECK(hipMalloc((float **)&d_input, nBytes));

	// transfer data from host to device
	CHECK(hipMemcpy(d_input, h_input, nBytes, hipMemcpyHostToDevice));

	// invoke kernel at host side
	int iLen = 512;
	dim3 block(iLen);
	dim3 grid((nElem + block.x - 1) / block.x);
	hipEvent_t start, stop;
	CHECK(hipEventCreate(&start));
	CHECK(hipEventCreate(&stop));

	for (int t = 0; t < TIMES; t++)
	{
	// record start event
		CHECK(hipEventRecord(start, 0));
#if defined(_2TREE)
		RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, d_vR);
#elif defined(_3TREE)
		RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, d_vR);
#elif defined(_4TREE)
		RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, d_vR);
#elif defined(_5TREE)
		RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, 0, d_vR);
#elif defined(_6TREE)
		RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, 0, 0, d_vR);
#elif defined(_7TREE)
		RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, 0, 0, 0, d_vR);
#elif defined(_8TREE)
		RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, 0, 0, 0, 0, d_vR);
#elif defined(_9TREE)
		RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, 0, 0, 0, 0, 0, d_vR);
#elif defined(_10TREE)
		RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, d_vR);
#else
		RF_kernel<<<grid, block>>>(d_input, nElem, 0, d_vR);
#endif
		CHECK(hipEventRecord(stop, 0));
		CHECK(hipEventSynchronize(stop));
		hipDeviceSynchronize();

		// calculate elapsed time
		CHECK(hipEventElapsedTime(&elapsed_time[t], start, stop));
		printf("[CUDA - Random Forest]: %s - execution time = %.6fms\n", EXP_NAME, elapsed_time[t]);
		registerTime(elapsed_time[t]);

		avg += elapsed_time[t];
	}
	// copy kernel result back to host side
	CHECK(hipMemcpy(h_vR, d_vR, nBytes, hipMemcpyDeviceToHost));
	CHECK(hipGetLastError());

	//Verify the answers
	char flag = 1;
	for (int i = 0; i < nElem; i++)
	{
		if (i < 32)
		{
			//printf("%.3f ", h_vR[i]);
		}
		if (h_input[i] != h_vR[i])
		{
			flag = 0;
			break;
		}
	}
	if (flag == 1)
	{
		printf("Return correct");
	}
	else
	{
		printf("Error!!");
	}
	printf("\n ");

	// avg and standard deviation
	avg = (float)avg / TIMES;
	for (int t = 0; t < TIMES; t++)
	{
		sd+=pow(elapsed_time[t]-avg,2);
	}
	sd = (float) sd/TIMES;
	sd = sqrt(sd);
	printf("\nAVG = %fms, SD = %fms\n", avg,sd);
	// free device memory
	hipFree(d_input);
	hipFree(d_vR);

	// free host memory
	free(h_input);
	free(h_vR);
	return (0);
}

void initData(float *ip, int size)
{
	for (int i = 0; i < size; i++)
	{

#if defined(_BEST)
		ip[i] = (float)(0);
#else
		if(WARP_SIZE < _MOD)
		{
			int aux = _MOD / WARP_SIZE;
			ip[i] = (float)((i * aux) % _MOD);
		}
		else
		{
			ip[i] = (float)(i % _MOD);
		}
#endif
	}
	return;
}

void registerTime(float value)
{
	printf("[CUDA - Random Forest]: Time spent: %f.\n", value);
}
