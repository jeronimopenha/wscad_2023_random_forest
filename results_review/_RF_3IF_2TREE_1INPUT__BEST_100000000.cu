#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>

#define _BEST
#define TIMES 10
#define N_ELEM 100000000

#define WARP_SIZE 32

#define CHECK(call)                                                \
	{                                                              \
		const hipError_t error = call;                            \
		if (error != hipSuccess)                                  \
		{                                                          \
			fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
			fprintf(stderr, "code: %d, reason: %s\n", error,       \
					hipGetErrorString(error));                    \
		}                                                          \
	}

void initData(float *ip, int size);
void registerTime(float value);

#define _MOD 4
#define _2TREE
#define EXP_NAME "_RF_3IF_2TREE_1INPUT__BEST_100000000"

// RF_with_if

__global__ void RF_kernel(float *input, const int n, const int offset0, const int offset1, float *vR)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
if(input[i]>(1+offset0))
{
	if(input[i]==(3+offset0))
{
	vR[i] = 3;
}
else
{
	vR[i] = 2;
}

}
else
{
	if(input[i]==(1+offset0))
{
	vR[i] = 1;
}
else
{
	vR[i] = 0;
}

}


if(input[i]>(1+offset1))
{
	if(input[i]==(3+offset1))
{
	vR[i] = 3;
}
else
{
	vR[i] = 2;
}

}
else
{
	if(input[i]==(1+offset1))
{
	vR[i] = 1;
}
else
{
	vR[i] = 0;
}

}


	}
}

int main(int argc, char **argv)
{
	float elapsed_time[TIMES], avg = 0.0, sd = 0.0;
	// set up device
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("[CUDA - Random Forest]: EXP %s\n", EXP_NAME);
	printf("[CUDA - Random Forest]: Using Device %d: %s\n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));

	// set up data size of vectors
	int nElem = N_ELEM;
	printf("[CUDA - Random Forest]: Vector Size %d\n", nElem);

	// malloc host memory
	size_t nBytes = nElem * sizeof(float);
	float *h_vR;
	h_vR = (float *)malloc(nBytes);
	float *h_input;
	h_input = (float *)malloc(nBytes);

	printf("[CUDA - Random Forest]: Start Reading Dataset\n");
	initData(h_input, nElem);
	printf("[CUDA - Random Forest]: Finished Reading Dataset\n");
	memset(h_vR, 0, nBytes);

	// malloc device global memory
	float *d_vR;
	CHECK(hipMalloc((float **)&d_vR, nBytes));
	float *d_input;
	CHECK(hipMalloc((float **)&d_input, nBytes));

	// transfer data from host to device
	CHECK(hipMemcpy(d_input, h_input, nBytes, hipMemcpyHostToDevice));

	// invoke kernel at host side
	int iLen = 512;
	dim3 block(iLen);
	dim3 grid((nElem + block.x - 1) / block.x);
	hipEvent_t start, stop;
	CHECK(hipEventCreate(&start));
	CHECK(hipEventCreate(&stop));

	for (int t = 0; t < TIMES; t++)
	{
	// record start event
		CHECK(hipEventRecord(start, 0));
#if defined(_2TREE)
		RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, d_vR);
#elif defined(_3TREE)
		RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, d_vR);
#elif defined(_4TREE)
		RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, d_vR);
#elif defined(_5TREE)
		RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, 0, d_vR);
#elif defined(_6TREE)
		RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, 0, 0, d_vR);
#elif defined(_7TREE)
		RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, 0, 0, 0, d_vR);
#elif defined(_8TREE)
		RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, 0, 0, 0, 0, d_vR);
#elif defined(_9TREE)
		RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, 0, 0, 0, 0, 0, d_vR);
#elif defined(_10TREE)
		RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, d_vR);
#else
		RF_kernel<<<grid, block>>>(d_input, nElem, 0, d_vR);
#endif
		CHECK(hipEventRecord(stop, 0));
		CHECK(hipEventSynchronize(stop));
		hipDeviceSynchronize();

		// calculate elapsed time
		CHECK(hipEventElapsedTime(&elapsed_time[t], start, stop));
		printf("[CUDA - Random Forest]: %s - execution time = %.6fms\n", EXP_NAME, elapsed_time[t]);
		registerTime(elapsed_time[t]);

		avg += elapsed_time[t];
	}
	// copy kernel result back to host side
	CHECK(hipMemcpy(h_vR, d_vR, nBytes, hipMemcpyDeviceToHost));
	CHECK(hipGetLastError());

	//Verify the answers
	char flag = 1;
	for (int i = 0; i < nElem; i++)
	{
		if (i < 32)
		{
			//printf("%.3f ", h_vR[i]);
		}
		if (h_input[i] != h_vR[i])
		{
			flag = 0;
			break;
		}
	}
	if (flag == 1)
	{
		printf("Return correct");
	}
	else
	{
		printf("Error!!");
	}
	printf("\n ");

	// avg and standard deviation
	avg = (float)avg / TIMES;
	for (int t = 0; t < TIMES; t++)
	{
		sd+=pow(elapsed_time[t]-avg,2);
	}
	sd = (float) sd/TIMES;
	sd = sqrt(sd);
	printf("\nAVG = %fms, SD = %fms\n", avg,sd);
	// free device memory
	hipFree(d_input);
	hipFree(d_vR);

	// free host memory
	free(h_input);
	free(h_vR);
	return (0);
}

void initData(float *ip, int size)
{
	for (int i = 0; i < size; i++)
	{

#if defined(_BEST)
		ip[i] = (float)(0);
#else
		if(WARP_SIZE < _MOD)
		{
			int aux = _MOD / WARP_SIZE;
			ip[i] = (float)((i * aux) % _MOD);
		}
		else
		{
			ip[i] = (float)(i % _MOD);
		}
#endif
	}
	return;
}

void registerTime(float value)
{
	printf("[CUDA - Random Forest]: Time spent: %f.\n", value);
}
