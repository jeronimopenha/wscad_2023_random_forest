#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>

#define _BEST
#define TIMES 10
#define N_ELEM 100000000

#define WARP_SIZE 32

#define CHECK(call)                                                \
	{                                                              \
		const hipError_t error = call;                            \
		if (error != hipSuccess)                                  \
		{                                                          \
			fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
			fprintf(stderr, "code: %d, reason: %s\n", error,       \
					hipGetErrorString(error));                    \
		}                                                          \
	}

void initData(float *ip, int size);
void registerTime(float value);

#define _MOD 4096
#define _1TREE
#define EXP_NAME "_RF_4095IF_1TREE_1INPUT__BEST_100000000"

// RF_with_if

__global__ void RF_kernel(float *input, const int n, const int offset0, float *vR)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
if(input[i]>(2047+offset0))
{
	if(input[i]>(3071+offset0))
{
	if(input[i]>(3583+offset0))
{
	if(input[i]>(3839+offset0))
{
	if(input[i]>(3967+offset0))
{
	if(input[i]>(4031+offset0))
{
	if(input[i]>(4063+offset0))
{
	if(input[i]>(4079+offset0))
{
	if(input[i]>(4087+offset0))
{
	if(input[i]>(4091+offset0))
{
	if(input[i]>(4093+offset0))
{
	if(input[i]==(4095+offset0))
{
	vR[i] = 4095;
}
else
{
	vR[i] = 4094;
}

}
else
{
	if(input[i]==(4093+offset0))
{
	vR[i] = 4093;
}
else
{
	vR[i] = 4092;
}

}

}
else
{
	if(input[i]>(4089+offset0))
{
	if(input[i]==(4091+offset0))
{
	vR[i] = 4091;
}
else
{
	vR[i] = 4090;
}

}
else
{
	if(input[i]==(4089+offset0))
{
	vR[i] = 4089;
}
else
{
	vR[i] = 4088;
}

}

}

}
else
{
	if(input[i]>(4083+offset0))
{
	if(input[i]>(4085+offset0))
{
	if(input[i]==(4087+offset0))
{
	vR[i] = 4087;
}
else
{
	vR[i] = 4086;
}

}
else
{
	if(input[i]==(4085+offset0))
{
	vR[i] = 4085;
}
else
{
	vR[i] = 4084;
}

}

}
else
{
	if(input[i]>(4081+offset0))
{
	if(input[i]==(4083+offset0))
{
	vR[i] = 4083;
}
else
{
	vR[i] = 4082;
}

}
else
{
	if(input[i]==(4081+offset0))
{
	vR[i] = 4081;
}
else
{
	vR[i] = 4080;
}

}

}

}

}
else
{
	if(input[i]>(4071+offset0))
{
	if(input[i]>(4075+offset0))
{
	if(input[i]>(4077+offset0))
{
	if(input[i]==(4079+offset0))
{
	vR[i] = 4079;
}
else
{
	vR[i] = 4078;
}

}
else
{
	if(input[i]==(4077+offset0))
{
	vR[i] = 4077;
}
else
{
	vR[i] = 4076;
}

}

}
else
{
	if(input[i]>(4073+offset0))
{
	if(input[i]==(4075+offset0))
{
	vR[i] = 4075;
}
else
{
	vR[i] = 4074;
}

}
else
{
	if(input[i]==(4073+offset0))
{
	vR[i] = 4073;
}
else
{
	vR[i] = 4072;
}

}

}

}
else
{
	if(input[i]>(4067+offset0))
{
	if(input[i]>(4069+offset0))
{
	if(input[i]==(4071+offset0))
{
	vR[i] = 4071;
}
else
{
	vR[i] = 4070;
}

}
else
{
	if(input[i]==(4069+offset0))
{
	vR[i] = 4069;
}
else
{
	vR[i] = 4068;
}

}

}
else
{
	if(input[i]>(4065+offset0))
{
	if(input[i]==(4067+offset0))
{
	vR[i] = 4067;
}
else
{
	vR[i] = 4066;
}

}
else
{
	if(input[i]==(4065+offset0))
{
	vR[i] = 4065;
}
else
{
	vR[i] = 4064;
}

}

}

}

}

}
else
{
	if(input[i]>(4047+offset0))
{
	if(input[i]>(4055+offset0))
{
	if(input[i]>(4059+offset0))
{
	if(input[i]>(4061+offset0))
{
	if(input[i]==(4063+offset0))
{
	vR[i] = 4063;
}
else
{
	vR[i] = 4062;
}

}
else
{
	if(input[i]==(4061+offset0))
{
	vR[i] = 4061;
}
else
{
	vR[i] = 4060;
}

}

}
else
{
	if(input[i]>(4057+offset0))
{
	if(input[i]==(4059+offset0))
{
	vR[i] = 4059;
}
else
{
	vR[i] = 4058;
}

}
else
{
	if(input[i]==(4057+offset0))
{
	vR[i] = 4057;
}
else
{
	vR[i] = 4056;
}

}

}

}
else
{
	if(input[i]>(4051+offset0))
{
	if(input[i]>(4053+offset0))
{
	if(input[i]==(4055+offset0))
{
	vR[i] = 4055;
}
else
{
	vR[i] = 4054;
}

}
else
{
	if(input[i]==(4053+offset0))
{
	vR[i] = 4053;
}
else
{
	vR[i] = 4052;
}

}

}
else
{
	if(input[i]>(4049+offset0))
{
	if(input[i]==(4051+offset0))
{
	vR[i] = 4051;
}
else
{
	vR[i] = 4050;
}

}
else
{
	if(input[i]==(4049+offset0))
{
	vR[i] = 4049;
}
else
{
	vR[i] = 4048;
}

}

}

}

}
else
{
	if(input[i]>(4039+offset0))
{
	if(input[i]>(4043+offset0))
{
	if(input[i]>(4045+offset0))
{
	if(input[i]==(4047+offset0))
{
	vR[i] = 4047;
}
else
{
	vR[i] = 4046;
}

}
else
{
	if(input[i]==(4045+offset0))
{
	vR[i] = 4045;
}
else
{
	vR[i] = 4044;
}

}

}
else
{
	if(input[i]>(4041+offset0))
{
	if(input[i]==(4043+offset0))
{
	vR[i] = 4043;
}
else
{
	vR[i] = 4042;
}

}
else
{
	if(input[i]==(4041+offset0))
{
	vR[i] = 4041;
}
else
{
	vR[i] = 4040;
}

}

}

}
else
{
	if(input[i]>(4035+offset0))
{
	if(input[i]>(4037+offset0))
{
	if(input[i]==(4039+offset0))
{
	vR[i] = 4039;
}
else
{
	vR[i] = 4038;
}

}
else
{
	if(input[i]==(4037+offset0))
{
	vR[i] = 4037;
}
else
{
	vR[i] = 4036;
}

}

}
else
{
	if(input[i]>(4033+offset0))
{
	if(input[i]==(4035+offset0))
{
	vR[i] = 4035;
}
else
{
	vR[i] = 4034;
}

}
else
{
	if(input[i]==(4033+offset0))
{
	vR[i] = 4033;
}
else
{
	vR[i] = 4032;
}

}

}

}

}

}

}
else
{
	if(input[i]>(3999+offset0))
{
	if(input[i]>(4015+offset0))
{
	if(input[i]>(4023+offset0))
{
	if(input[i]>(4027+offset0))
{
	if(input[i]>(4029+offset0))
{
	if(input[i]==(4031+offset0))
{
	vR[i] = 4031;
}
else
{
	vR[i] = 4030;
}

}
else
{
	if(input[i]==(4029+offset0))
{
	vR[i] = 4029;
}
else
{
	vR[i] = 4028;
}

}

}
else
{
	if(input[i]>(4025+offset0))
{
	if(input[i]==(4027+offset0))
{
	vR[i] = 4027;
}
else
{
	vR[i] = 4026;
}

}
else
{
	if(input[i]==(4025+offset0))
{
	vR[i] = 4025;
}
else
{
	vR[i] = 4024;
}

}

}

}
else
{
	if(input[i]>(4019+offset0))
{
	if(input[i]>(4021+offset0))
{
	if(input[i]==(4023+offset0))
{
	vR[i] = 4023;
}
else
{
	vR[i] = 4022;
}

}
else
{
	if(input[i]==(4021+offset0))
{
	vR[i] = 4021;
}
else
{
	vR[i] = 4020;
}

}

}
else
{
	if(input[i]>(4017+offset0))
{
	if(input[i]==(4019+offset0))
{
	vR[i] = 4019;
}
else
{
	vR[i] = 4018;
}

}
else
{
	if(input[i]==(4017+offset0))
{
	vR[i] = 4017;
}
else
{
	vR[i] = 4016;
}

}

}

}

}
else
{
	if(input[i]>(4007+offset0))
{
	if(input[i]>(4011+offset0))
{
	if(input[i]>(4013+offset0))
{
	if(input[i]==(4015+offset0))
{
	vR[i] = 4015;
}
else
{
	vR[i] = 4014;
}

}
else
{
	if(input[i]==(4013+offset0))
{
	vR[i] = 4013;
}
else
{
	vR[i] = 4012;
}

}

}
else
{
	if(input[i]>(4009+offset0))
{
	if(input[i]==(4011+offset0))
{
	vR[i] = 4011;
}
else
{
	vR[i] = 4010;
}

}
else
{
	if(input[i]==(4009+offset0))
{
	vR[i] = 4009;
}
else
{
	vR[i] = 4008;
}

}

}

}
else
{
	if(input[i]>(4003+offset0))
{
	if(input[i]>(4005+offset0))
{
	if(input[i]==(4007+offset0))
{
	vR[i] = 4007;
}
else
{
	vR[i] = 4006;
}

}
else
{
	if(input[i]==(4005+offset0))
{
	vR[i] = 4005;
}
else
{
	vR[i] = 4004;
}

}

}
else
{
	if(input[i]>(4001+offset0))
{
	if(input[i]==(4003+offset0))
{
	vR[i] = 4003;
}
else
{
	vR[i] = 4002;
}

}
else
{
	if(input[i]==(4001+offset0))
{
	vR[i] = 4001;
}
else
{
	vR[i] = 4000;
}

}

}

}

}

}
else
{
	if(input[i]>(3983+offset0))
{
	if(input[i]>(3991+offset0))
{
	if(input[i]>(3995+offset0))
{
	if(input[i]>(3997+offset0))
{
	if(input[i]==(3999+offset0))
{
	vR[i] = 3999;
}
else
{
	vR[i] = 3998;
}

}
else
{
	if(input[i]==(3997+offset0))
{
	vR[i] = 3997;
}
else
{
	vR[i] = 3996;
}

}

}
else
{
	if(input[i]>(3993+offset0))
{
	if(input[i]==(3995+offset0))
{
	vR[i] = 3995;
}
else
{
	vR[i] = 3994;
}

}
else
{
	if(input[i]==(3993+offset0))
{
	vR[i] = 3993;
}
else
{
	vR[i] = 3992;
}

}

}

}
else
{
	if(input[i]>(3987+offset0))
{
	if(input[i]>(3989+offset0))
{
	if(input[i]==(3991+offset0))
{
	vR[i] = 3991;
}
else
{
	vR[i] = 3990;
}

}
else
{
	if(input[i]==(3989+offset0))
{
	vR[i] = 3989;
}
else
{
	vR[i] = 3988;
}

}

}
else
{
	if(input[i]>(3985+offset0))
{
	if(input[i]==(3987+offset0))
{
	vR[i] = 3987;
}
else
{
	vR[i] = 3986;
}

}
else
{
	if(input[i]==(3985+offset0))
{
	vR[i] = 3985;
}
else
{
	vR[i] = 3984;
}

}

}

}

}
else
{
	if(input[i]>(3975+offset0))
{
	if(input[i]>(3979+offset0))
{
	if(input[i]>(3981+offset0))
{
	if(input[i]==(3983+offset0))
{
	vR[i] = 3983;
}
else
{
	vR[i] = 3982;
}

}
else
{
	if(input[i]==(3981+offset0))
{
	vR[i] = 3981;
}
else
{
	vR[i] = 3980;
}

}

}
else
{
	if(input[i]>(3977+offset0))
{
	if(input[i]==(3979+offset0))
{
	vR[i] = 3979;
}
else
{
	vR[i] = 3978;
}

}
else
{
	if(input[i]==(3977+offset0))
{
	vR[i] = 3977;
}
else
{
	vR[i] = 3976;
}

}

}

}
else
{
	if(input[i]>(3971+offset0))
{
	if(input[i]>(3973+offset0))
{
	if(input[i]==(3975+offset0))
{
	vR[i] = 3975;
}
else
{
	vR[i] = 3974;
}

}
else
{
	if(input[i]==(3973+offset0))
{
	vR[i] = 3973;
}
else
{
	vR[i] = 3972;
}

}

}
else
{
	if(input[i]>(3969+offset0))
{
	if(input[i]==(3971+offset0))
{
	vR[i] = 3971;
}
else
{
	vR[i] = 3970;
}

}
else
{
	if(input[i]==(3969+offset0))
{
	vR[i] = 3969;
}
else
{
	vR[i] = 3968;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(3903+offset0))
{
	if(input[i]>(3935+offset0))
{
	if(input[i]>(3951+offset0))
{
	if(input[i]>(3959+offset0))
{
	if(input[i]>(3963+offset0))
{
	if(input[i]>(3965+offset0))
{
	if(input[i]==(3967+offset0))
{
	vR[i] = 3967;
}
else
{
	vR[i] = 3966;
}

}
else
{
	if(input[i]==(3965+offset0))
{
	vR[i] = 3965;
}
else
{
	vR[i] = 3964;
}

}

}
else
{
	if(input[i]>(3961+offset0))
{
	if(input[i]==(3963+offset0))
{
	vR[i] = 3963;
}
else
{
	vR[i] = 3962;
}

}
else
{
	if(input[i]==(3961+offset0))
{
	vR[i] = 3961;
}
else
{
	vR[i] = 3960;
}

}

}

}
else
{
	if(input[i]>(3955+offset0))
{
	if(input[i]>(3957+offset0))
{
	if(input[i]==(3959+offset0))
{
	vR[i] = 3959;
}
else
{
	vR[i] = 3958;
}

}
else
{
	if(input[i]==(3957+offset0))
{
	vR[i] = 3957;
}
else
{
	vR[i] = 3956;
}

}

}
else
{
	if(input[i]>(3953+offset0))
{
	if(input[i]==(3955+offset0))
{
	vR[i] = 3955;
}
else
{
	vR[i] = 3954;
}

}
else
{
	if(input[i]==(3953+offset0))
{
	vR[i] = 3953;
}
else
{
	vR[i] = 3952;
}

}

}

}

}
else
{
	if(input[i]>(3943+offset0))
{
	if(input[i]>(3947+offset0))
{
	if(input[i]>(3949+offset0))
{
	if(input[i]==(3951+offset0))
{
	vR[i] = 3951;
}
else
{
	vR[i] = 3950;
}

}
else
{
	if(input[i]==(3949+offset0))
{
	vR[i] = 3949;
}
else
{
	vR[i] = 3948;
}

}

}
else
{
	if(input[i]>(3945+offset0))
{
	if(input[i]==(3947+offset0))
{
	vR[i] = 3947;
}
else
{
	vR[i] = 3946;
}

}
else
{
	if(input[i]==(3945+offset0))
{
	vR[i] = 3945;
}
else
{
	vR[i] = 3944;
}

}

}

}
else
{
	if(input[i]>(3939+offset0))
{
	if(input[i]>(3941+offset0))
{
	if(input[i]==(3943+offset0))
{
	vR[i] = 3943;
}
else
{
	vR[i] = 3942;
}

}
else
{
	if(input[i]==(3941+offset0))
{
	vR[i] = 3941;
}
else
{
	vR[i] = 3940;
}

}

}
else
{
	if(input[i]>(3937+offset0))
{
	if(input[i]==(3939+offset0))
{
	vR[i] = 3939;
}
else
{
	vR[i] = 3938;
}

}
else
{
	if(input[i]==(3937+offset0))
{
	vR[i] = 3937;
}
else
{
	vR[i] = 3936;
}

}

}

}

}

}
else
{
	if(input[i]>(3919+offset0))
{
	if(input[i]>(3927+offset0))
{
	if(input[i]>(3931+offset0))
{
	if(input[i]>(3933+offset0))
{
	if(input[i]==(3935+offset0))
{
	vR[i] = 3935;
}
else
{
	vR[i] = 3934;
}

}
else
{
	if(input[i]==(3933+offset0))
{
	vR[i] = 3933;
}
else
{
	vR[i] = 3932;
}

}

}
else
{
	if(input[i]>(3929+offset0))
{
	if(input[i]==(3931+offset0))
{
	vR[i] = 3931;
}
else
{
	vR[i] = 3930;
}

}
else
{
	if(input[i]==(3929+offset0))
{
	vR[i] = 3929;
}
else
{
	vR[i] = 3928;
}

}

}

}
else
{
	if(input[i]>(3923+offset0))
{
	if(input[i]>(3925+offset0))
{
	if(input[i]==(3927+offset0))
{
	vR[i] = 3927;
}
else
{
	vR[i] = 3926;
}

}
else
{
	if(input[i]==(3925+offset0))
{
	vR[i] = 3925;
}
else
{
	vR[i] = 3924;
}

}

}
else
{
	if(input[i]>(3921+offset0))
{
	if(input[i]==(3923+offset0))
{
	vR[i] = 3923;
}
else
{
	vR[i] = 3922;
}

}
else
{
	if(input[i]==(3921+offset0))
{
	vR[i] = 3921;
}
else
{
	vR[i] = 3920;
}

}

}

}

}
else
{
	if(input[i]>(3911+offset0))
{
	if(input[i]>(3915+offset0))
{
	if(input[i]>(3917+offset0))
{
	if(input[i]==(3919+offset0))
{
	vR[i] = 3919;
}
else
{
	vR[i] = 3918;
}

}
else
{
	if(input[i]==(3917+offset0))
{
	vR[i] = 3917;
}
else
{
	vR[i] = 3916;
}

}

}
else
{
	if(input[i]>(3913+offset0))
{
	if(input[i]==(3915+offset0))
{
	vR[i] = 3915;
}
else
{
	vR[i] = 3914;
}

}
else
{
	if(input[i]==(3913+offset0))
{
	vR[i] = 3913;
}
else
{
	vR[i] = 3912;
}

}

}

}
else
{
	if(input[i]>(3907+offset0))
{
	if(input[i]>(3909+offset0))
{
	if(input[i]==(3911+offset0))
{
	vR[i] = 3911;
}
else
{
	vR[i] = 3910;
}

}
else
{
	if(input[i]==(3909+offset0))
{
	vR[i] = 3909;
}
else
{
	vR[i] = 3908;
}

}

}
else
{
	if(input[i]>(3905+offset0))
{
	if(input[i]==(3907+offset0))
{
	vR[i] = 3907;
}
else
{
	vR[i] = 3906;
}

}
else
{
	if(input[i]==(3905+offset0))
{
	vR[i] = 3905;
}
else
{
	vR[i] = 3904;
}

}

}

}

}

}

}
else
{
	if(input[i]>(3871+offset0))
{
	if(input[i]>(3887+offset0))
{
	if(input[i]>(3895+offset0))
{
	if(input[i]>(3899+offset0))
{
	if(input[i]>(3901+offset0))
{
	if(input[i]==(3903+offset0))
{
	vR[i] = 3903;
}
else
{
	vR[i] = 3902;
}

}
else
{
	if(input[i]==(3901+offset0))
{
	vR[i] = 3901;
}
else
{
	vR[i] = 3900;
}

}

}
else
{
	if(input[i]>(3897+offset0))
{
	if(input[i]==(3899+offset0))
{
	vR[i] = 3899;
}
else
{
	vR[i] = 3898;
}

}
else
{
	if(input[i]==(3897+offset0))
{
	vR[i] = 3897;
}
else
{
	vR[i] = 3896;
}

}

}

}
else
{
	if(input[i]>(3891+offset0))
{
	if(input[i]>(3893+offset0))
{
	if(input[i]==(3895+offset0))
{
	vR[i] = 3895;
}
else
{
	vR[i] = 3894;
}

}
else
{
	if(input[i]==(3893+offset0))
{
	vR[i] = 3893;
}
else
{
	vR[i] = 3892;
}

}

}
else
{
	if(input[i]>(3889+offset0))
{
	if(input[i]==(3891+offset0))
{
	vR[i] = 3891;
}
else
{
	vR[i] = 3890;
}

}
else
{
	if(input[i]==(3889+offset0))
{
	vR[i] = 3889;
}
else
{
	vR[i] = 3888;
}

}

}

}

}
else
{
	if(input[i]>(3879+offset0))
{
	if(input[i]>(3883+offset0))
{
	if(input[i]>(3885+offset0))
{
	if(input[i]==(3887+offset0))
{
	vR[i] = 3887;
}
else
{
	vR[i] = 3886;
}

}
else
{
	if(input[i]==(3885+offset0))
{
	vR[i] = 3885;
}
else
{
	vR[i] = 3884;
}

}

}
else
{
	if(input[i]>(3881+offset0))
{
	if(input[i]==(3883+offset0))
{
	vR[i] = 3883;
}
else
{
	vR[i] = 3882;
}

}
else
{
	if(input[i]==(3881+offset0))
{
	vR[i] = 3881;
}
else
{
	vR[i] = 3880;
}

}

}

}
else
{
	if(input[i]>(3875+offset0))
{
	if(input[i]>(3877+offset0))
{
	if(input[i]==(3879+offset0))
{
	vR[i] = 3879;
}
else
{
	vR[i] = 3878;
}

}
else
{
	if(input[i]==(3877+offset0))
{
	vR[i] = 3877;
}
else
{
	vR[i] = 3876;
}

}

}
else
{
	if(input[i]>(3873+offset0))
{
	if(input[i]==(3875+offset0))
{
	vR[i] = 3875;
}
else
{
	vR[i] = 3874;
}

}
else
{
	if(input[i]==(3873+offset0))
{
	vR[i] = 3873;
}
else
{
	vR[i] = 3872;
}

}

}

}

}

}
else
{
	if(input[i]>(3855+offset0))
{
	if(input[i]>(3863+offset0))
{
	if(input[i]>(3867+offset0))
{
	if(input[i]>(3869+offset0))
{
	if(input[i]==(3871+offset0))
{
	vR[i] = 3871;
}
else
{
	vR[i] = 3870;
}

}
else
{
	if(input[i]==(3869+offset0))
{
	vR[i] = 3869;
}
else
{
	vR[i] = 3868;
}

}

}
else
{
	if(input[i]>(3865+offset0))
{
	if(input[i]==(3867+offset0))
{
	vR[i] = 3867;
}
else
{
	vR[i] = 3866;
}

}
else
{
	if(input[i]==(3865+offset0))
{
	vR[i] = 3865;
}
else
{
	vR[i] = 3864;
}

}

}

}
else
{
	if(input[i]>(3859+offset0))
{
	if(input[i]>(3861+offset0))
{
	if(input[i]==(3863+offset0))
{
	vR[i] = 3863;
}
else
{
	vR[i] = 3862;
}

}
else
{
	if(input[i]==(3861+offset0))
{
	vR[i] = 3861;
}
else
{
	vR[i] = 3860;
}

}

}
else
{
	if(input[i]>(3857+offset0))
{
	if(input[i]==(3859+offset0))
{
	vR[i] = 3859;
}
else
{
	vR[i] = 3858;
}

}
else
{
	if(input[i]==(3857+offset0))
{
	vR[i] = 3857;
}
else
{
	vR[i] = 3856;
}

}

}

}

}
else
{
	if(input[i]>(3847+offset0))
{
	if(input[i]>(3851+offset0))
{
	if(input[i]>(3853+offset0))
{
	if(input[i]==(3855+offset0))
{
	vR[i] = 3855;
}
else
{
	vR[i] = 3854;
}

}
else
{
	if(input[i]==(3853+offset0))
{
	vR[i] = 3853;
}
else
{
	vR[i] = 3852;
}

}

}
else
{
	if(input[i]>(3849+offset0))
{
	if(input[i]==(3851+offset0))
{
	vR[i] = 3851;
}
else
{
	vR[i] = 3850;
}

}
else
{
	if(input[i]==(3849+offset0))
{
	vR[i] = 3849;
}
else
{
	vR[i] = 3848;
}

}

}

}
else
{
	if(input[i]>(3843+offset0))
{
	if(input[i]>(3845+offset0))
{
	if(input[i]==(3847+offset0))
{
	vR[i] = 3847;
}
else
{
	vR[i] = 3846;
}

}
else
{
	if(input[i]==(3845+offset0))
{
	vR[i] = 3845;
}
else
{
	vR[i] = 3844;
}

}

}
else
{
	if(input[i]>(3841+offset0))
{
	if(input[i]==(3843+offset0))
{
	vR[i] = 3843;
}
else
{
	vR[i] = 3842;
}

}
else
{
	if(input[i]==(3841+offset0))
{
	vR[i] = 3841;
}
else
{
	vR[i] = 3840;
}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(3711+offset0))
{
	if(input[i]>(3775+offset0))
{
	if(input[i]>(3807+offset0))
{
	if(input[i]>(3823+offset0))
{
	if(input[i]>(3831+offset0))
{
	if(input[i]>(3835+offset0))
{
	if(input[i]>(3837+offset0))
{
	if(input[i]==(3839+offset0))
{
	vR[i] = 3839;
}
else
{
	vR[i] = 3838;
}

}
else
{
	if(input[i]==(3837+offset0))
{
	vR[i] = 3837;
}
else
{
	vR[i] = 3836;
}

}

}
else
{
	if(input[i]>(3833+offset0))
{
	if(input[i]==(3835+offset0))
{
	vR[i] = 3835;
}
else
{
	vR[i] = 3834;
}

}
else
{
	if(input[i]==(3833+offset0))
{
	vR[i] = 3833;
}
else
{
	vR[i] = 3832;
}

}

}

}
else
{
	if(input[i]>(3827+offset0))
{
	if(input[i]>(3829+offset0))
{
	if(input[i]==(3831+offset0))
{
	vR[i] = 3831;
}
else
{
	vR[i] = 3830;
}

}
else
{
	if(input[i]==(3829+offset0))
{
	vR[i] = 3829;
}
else
{
	vR[i] = 3828;
}

}

}
else
{
	if(input[i]>(3825+offset0))
{
	if(input[i]==(3827+offset0))
{
	vR[i] = 3827;
}
else
{
	vR[i] = 3826;
}

}
else
{
	if(input[i]==(3825+offset0))
{
	vR[i] = 3825;
}
else
{
	vR[i] = 3824;
}

}

}

}

}
else
{
	if(input[i]>(3815+offset0))
{
	if(input[i]>(3819+offset0))
{
	if(input[i]>(3821+offset0))
{
	if(input[i]==(3823+offset0))
{
	vR[i] = 3823;
}
else
{
	vR[i] = 3822;
}

}
else
{
	if(input[i]==(3821+offset0))
{
	vR[i] = 3821;
}
else
{
	vR[i] = 3820;
}

}

}
else
{
	if(input[i]>(3817+offset0))
{
	if(input[i]==(3819+offset0))
{
	vR[i] = 3819;
}
else
{
	vR[i] = 3818;
}

}
else
{
	if(input[i]==(3817+offset0))
{
	vR[i] = 3817;
}
else
{
	vR[i] = 3816;
}

}

}

}
else
{
	if(input[i]>(3811+offset0))
{
	if(input[i]>(3813+offset0))
{
	if(input[i]==(3815+offset0))
{
	vR[i] = 3815;
}
else
{
	vR[i] = 3814;
}

}
else
{
	if(input[i]==(3813+offset0))
{
	vR[i] = 3813;
}
else
{
	vR[i] = 3812;
}

}

}
else
{
	if(input[i]>(3809+offset0))
{
	if(input[i]==(3811+offset0))
{
	vR[i] = 3811;
}
else
{
	vR[i] = 3810;
}

}
else
{
	if(input[i]==(3809+offset0))
{
	vR[i] = 3809;
}
else
{
	vR[i] = 3808;
}

}

}

}

}

}
else
{
	if(input[i]>(3791+offset0))
{
	if(input[i]>(3799+offset0))
{
	if(input[i]>(3803+offset0))
{
	if(input[i]>(3805+offset0))
{
	if(input[i]==(3807+offset0))
{
	vR[i] = 3807;
}
else
{
	vR[i] = 3806;
}

}
else
{
	if(input[i]==(3805+offset0))
{
	vR[i] = 3805;
}
else
{
	vR[i] = 3804;
}

}

}
else
{
	if(input[i]>(3801+offset0))
{
	if(input[i]==(3803+offset0))
{
	vR[i] = 3803;
}
else
{
	vR[i] = 3802;
}

}
else
{
	if(input[i]==(3801+offset0))
{
	vR[i] = 3801;
}
else
{
	vR[i] = 3800;
}

}

}

}
else
{
	if(input[i]>(3795+offset0))
{
	if(input[i]>(3797+offset0))
{
	if(input[i]==(3799+offset0))
{
	vR[i] = 3799;
}
else
{
	vR[i] = 3798;
}

}
else
{
	if(input[i]==(3797+offset0))
{
	vR[i] = 3797;
}
else
{
	vR[i] = 3796;
}

}

}
else
{
	if(input[i]>(3793+offset0))
{
	if(input[i]==(3795+offset0))
{
	vR[i] = 3795;
}
else
{
	vR[i] = 3794;
}

}
else
{
	if(input[i]==(3793+offset0))
{
	vR[i] = 3793;
}
else
{
	vR[i] = 3792;
}

}

}

}

}
else
{
	if(input[i]>(3783+offset0))
{
	if(input[i]>(3787+offset0))
{
	if(input[i]>(3789+offset0))
{
	if(input[i]==(3791+offset0))
{
	vR[i] = 3791;
}
else
{
	vR[i] = 3790;
}

}
else
{
	if(input[i]==(3789+offset0))
{
	vR[i] = 3789;
}
else
{
	vR[i] = 3788;
}

}

}
else
{
	if(input[i]>(3785+offset0))
{
	if(input[i]==(3787+offset0))
{
	vR[i] = 3787;
}
else
{
	vR[i] = 3786;
}

}
else
{
	if(input[i]==(3785+offset0))
{
	vR[i] = 3785;
}
else
{
	vR[i] = 3784;
}

}

}

}
else
{
	if(input[i]>(3779+offset0))
{
	if(input[i]>(3781+offset0))
{
	if(input[i]==(3783+offset0))
{
	vR[i] = 3783;
}
else
{
	vR[i] = 3782;
}

}
else
{
	if(input[i]==(3781+offset0))
{
	vR[i] = 3781;
}
else
{
	vR[i] = 3780;
}

}

}
else
{
	if(input[i]>(3777+offset0))
{
	if(input[i]==(3779+offset0))
{
	vR[i] = 3779;
}
else
{
	vR[i] = 3778;
}

}
else
{
	if(input[i]==(3777+offset0))
{
	vR[i] = 3777;
}
else
{
	vR[i] = 3776;
}

}

}

}

}

}

}
else
{
	if(input[i]>(3743+offset0))
{
	if(input[i]>(3759+offset0))
{
	if(input[i]>(3767+offset0))
{
	if(input[i]>(3771+offset0))
{
	if(input[i]>(3773+offset0))
{
	if(input[i]==(3775+offset0))
{
	vR[i] = 3775;
}
else
{
	vR[i] = 3774;
}

}
else
{
	if(input[i]==(3773+offset0))
{
	vR[i] = 3773;
}
else
{
	vR[i] = 3772;
}

}

}
else
{
	if(input[i]>(3769+offset0))
{
	if(input[i]==(3771+offset0))
{
	vR[i] = 3771;
}
else
{
	vR[i] = 3770;
}

}
else
{
	if(input[i]==(3769+offset0))
{
	vR[i] = 3769;
}
else
{
	vR[i] = 3768;
}

}

}

}
else
{
	if(input[i]>(3763+offset0))
{
	if(input[i]>(3765+offset0))
{
	if(input[i]==(3767+offset0))
{
	vR[i] = 3767;
}
else
{
	vR[i] = 3766;
}

}
else
{
	if(input[i]==(3765+offset0))
{
	vR[i] = 3765;
}
else
{
	vR[i] = 3764;
}

}

}
else
{
	if(input[i]>(3761+offset0))
{
	if(input[i]==(3763+offset0))
{
	vR[i] = 3763;
}
else
{
	vR[i] = 3762;
}

}
else
{
	if(input[i]==(3761+offset0))
{
	vR[i] = 3761;
}
else
{
	vR[i] = 3760;
}

}

}

}

}
else
{
	if(input[i]>(3751+offset0))
{
	if(input[i]>(3755+offset0))
{
	if(input[i]>(3757+offset0))
{
	if(input[i]==(3759+offset0))
{
	vR[i] = 3759;
}
else
{
	vR[i] = 3758;
}

}
else
{
	if(input[i]==(3757+offset0))
{
	vR[i] = 3757;
}
else
{
	vR[i] = 3756;
}

}

}
else
{
	if(input[i]>(3753+offset0))
{
	if(input[i]==(3755+offset0))
{
	vR[i] = 3755;
}
else
{
	vR[i] = 3754;
}

}
else
{
	if(input[i]==(3753+offset0))
{
	vR[i] = 3753;
}
else
{
	vR[i] = 3752;
}

}

}

}
else
{
	if(input[i]>(3747+offset0))
{
	if(input[i]>(3749+offset0))
{
	if(input[i]==(3751+offset0))
{
	vR[i] = 3751;
}
else
{
	vR[i] = 3750;
}

}
else
{
	if(input[i]==(3749+offset0))
{
	vR[i] = 3749;
}
else
{
	vR[i] = 3748;
}

}

}
else
{
	if(input[i]>(3745+offset0))
{
	if(input[i]==(3747+offset0))
{
	vR[i] = 3747;
}
else
{
	vR[i] = 3746;
}

}
else
{
	if(input[i]==(3745+offset0))
{
	vR[i] = 3745;
}
else
{
	vR[i] = 3744;
}

}

}

}

}

}
else
{
	if(input[i]>(3727+offset0))
{
	if(input[i]>(3735+offset0))
{
	if(input[i]>(3739+offset0))
{
	if(input[i]>(3741+offset0))
{
	if(input[i]==(3743+offset0))
{
	vR[i] = 3743;
}
else
{
	vR[i] = 3742;
}

}
else
{
	if(input[i]==(3741+offset0))
{
	vR[i] = 3741;
}
else
{
	vR[i] = 3740;
}

}

}
else
{
	if(input[i]>(3737+offset0))
{
	if(input[i]==(3739+offset0))
{
	vR[i] = 3739;
}
else
{
	vR[i] = 3738;
}

}
else
{
	if(input[i]==(3737+offset0))
{
	vR[i] = 3737;
}
else
{
	vR[i] = 3736;
}

}

}

}
else
{
	if(input[i]>(3731+offset0))
{
	if(input[i]>(3733+offset0))
{
	if(input[i]==(3735+offset0))
{
	vR[i] = 3735;
}
else
{
	vR[i] = 3734;
}

}
else
{
	if(input[i]==(3733+offset0))
{
	vR[i] = 3733;
}
else
{
	vR[i] = 3732;
}

}

}
else
{
	if(input[i]>(3729+offset0))
{
	if(input[i]==(3731+offset0))
{
	vR[i] = 3731;
}
else
{
	vR[i] = 3730;
}

}
else
{
	if(input[i]==(3729+offset0))
{
	vR[i] = 3729;
}
else
{
	vR[i] = 3728;
}

}

}

}

}
else
{
	if(input[i]>(3719+offset0))
{
	if(input[i]>(3723+offset0))
{
	if(input[i]>(3725+offset0))
{
	if(input[i]==(3727+offset0))
{
	vR[i] = 3727;
}
else
{
	vR[i] = 3726;
}

}
else
{
	if(input[i]==(3725+offset0))
{
	vR[i] = 3725;
}
else
{
	vR[i] = 3724;
}

}

}
else
{
	if(input[i]>(3721+offset0))
{
	if(input[i]==(3723+offset0))
{
	vR[i] = 3723;
}
else
{
	vR[i] = 3722;
}

}
else
{
	if(input[i]==(3721+offset0))
{
	vR[i] = 3721;
}
else
{
	vR[i] = 3720;
}

}

}

}
else
{
	if(input[i]>(3715+offset0))
{
	if(input[i]>(3717+offset0))
{
	if(input[i]==(3719+offset0))
{
	vR[i] = 3719;
}
else
{
	vR[i] = 3718;
}

}
else
{
	if(input[i]==(3717+offset0))
{
	vR[i] = 3717;
}
else
{
	vR[i] = 3716;
}

}

}
else
{
	if(input[i]>(3713+offset0))
{
	if(input[i]==(3715+offset0))
{
	vR[i] = 3715;
}
else
{
	vR[i] = 3714;
}

}
else
{
	if(input[i]==(3713+offset0))
{
	vR[i] = 3713;
}
else
{
	vR[i] = 3712;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(3647+offset0))
{
	if(input[i]>(3679+offset0))
{
	if(input[i]>(3695+offset0))
{
	if(input[i]>(3703+offset0))
{
	if(input[i]>(3707+offset0))
{
	if(input[i]>(3709+offset0))
{
	if(input[i]==(3711+offset0))
{
	vR[i] = 3711;
}
else
{
	vR[i] = 3710;
}

}
else
{
	if(input[i]==(3709+offset0))
{
	vR[i] = 3709;
}
else
{
	vR[i] = 3708;
}

}

}
else
{
	if(input[i]>(3705+offset0))
{
	if(input[i]==(3707+offset0))
{
	vR[i] = 3707;
}
else
{
	vR[i] = 3706;
}

}
else
{
	if(input[i]==(3705+offset0))
{
	vR[i] = 3705;
}
else
{
	vR[i] = 3704;
}

}

}

}
else
{
	if(input[i]>(3699+offset0))
{
	if(input[i]>(3701+offset0))
{
	if(input[i]==(3703+offset0))
{
	vR[i] = 3703;
}
else
{
	vR[i] = 3702;
}

}
else
{
	if(input[i]==(3701+offset0))
{
	vR[i] = 3701;
}
else
{
	vR[i] = 3700;
}

}

}
else
{
	if(input[i]>(3697+offset0))
{
	if(input[i]==(3699+offset0))
{
	vR[i] = 3699;
}
else
{
	vR[i] = 3698;
}

}
else
{
	if(input[i]==(3697+offset0))
{
	vR[i] = 3697;
}
else
{
	vR[i] = 3696;
}

}

}

}

}
else
{
	if(input[i]>(3687+offset0))
{
	if(input[i]>(3691+offset0))
{
	if(input[i]>(3693+offset0))
{
	if(input[i]==(3695+offset0))
{
	vR[i] = 3695;
}
else
{
	vR[i] = 3694;
}

}
else
{
	if(input[i]==(3693+offset0))
{
	vR[i] = 3693;
}
else
{
	vR[i] = 3692;
}

}

}
else
{
	if(input[i]>(3689+offset0))
{
	if(input[i]==(3691+offset0))
{
	vR[i] = 3691;
}
else
{
	vR[i] = 3690;
}

}
else
{
	if(input[i]==(3689+offset0))
{
	vR[i] = 3689;
}
else
{
	vR[i] = 3688;
}

}

}

}
else
{
	if(input[i]>(3683+offset0))
{
	if(input[i]>(3685+offset0))
{
	if(input[i]==(3687+offset0))
{
	vR[i] = 3687;
}
else
{
	vR[i] = 3686;
}

}
else
{
	if(input[i]==(3685+offset0))
{
	vR[i] = 3685;
}
else
{
	vR[i] = 3684;
}

}

}
else
{
	if(input[i]>(3681+offset0))
{
	if(input[i]==(3683+offset0))
{
	vR[i] = 3683;
}
else
{
	vR[i] = 3682;
}

}
else
{
	if(input[i]==(3681+offset0))
{
	vR[i] = 3681;
}
else
{
	vR[i] = 3680;
}

}

}

}

}

}
else
{
	if(input[i]>(3663+offset0))
{
	if(input[i]>(3671+offset0))
{
	if(input[i]>(3675+offset0))
{
	if(input[i]>(3677+offset0))
{
	if(input[i]==(3679+offset0))
{
	vR[i] = 3679;
}
else
{
	vR[i] = 3678;
}

}
else
{
	if(input[i]==(3677+offset0))
{
	vR[i] = 3677;
}
else
{
	vR[i] = 3676;
}

}

}
else
{
	if(input[i]>(3673+offset0))
{
	if(input[i]==(3675+offset0))
{
	vR[i] = 3675;
}
else
{
	vR[i] = 3674;
}

}
else
{
	if(input[i]==(3673+offset0))
{
	vR[i] = 3673;
}
else
{
	vR[i] = 3672;
}

}

}

}
else
{
	if(input[i]>(3667+offset0))
{
	if(input[i]>(3669+offset0))
{
	if(input[i]==(3671+offset0))
{
	vR[i] = 3671;
}
else
{
	vR[i] = 3670;
}

}
else
{
	if(input[i]==(3669+offset0))
{
	vR[i] = 3669;
}
else
{
	vR[i] = 3668;
}

}

}
else
{
	if(input[i]>(3665+offset0))
{
	if(input[i]==(3667+offset0))
{
	vR[i] = 3667;
}
else
{
	vR[i] = 3666;
}

}
else
{
	if(input[i]==(3665+offset0))
{
	vR[i] = 3665;
}
else
{
	vR[i] = 3664;
}

}

}

}

}
else
{
	if(input[i]>(3655+offset0))
{
	if(input[i]>(3659+offset0))
{
	if(input[i]>(3661+offset0))
{
	if(input[i]==(3663+offset0))
{
	vR[i] = 3663;
}
else
{
	vR[i] = 3662;
}

}
else
{
	if(input[i]==(3661+offset0))
{
	vR[i] = 3661;
}
else
{
	vR[i] = 3660;
}

}

}
else
{
	if(input[i]>(3657+offset0))
{
	if(input[i]==(3659+offset0))
{
	vR[i] = 3659;
}
else
{
	vR[i] = 3658;
}

}
else
{
	if(input[i]==(3657+offset0))
{
	vR[i] = 3657;
}
else
{
	vR[i] = 3656;
}

}

}

}
else
{
	if(input[i]>(3651+offset0))
{
	if(input[i]>(3653+offset0))
{
	if(input[i]==(3655+offset0))
{
	vR[i] = 3655;
}
else
{
	vR[i] = 3654;
}

}
else
{
	if(input[i]==(3653+offset0))
{
	vR[i] = 3653;
}
else
{
	vR[i] = 3652;
}

}

}
else
{
	if(input[i]>(3649+offset0))
{
	if(input[i]==(3651+offset0))
{
	vR[i] = 3651;
}
else
{
	vR[i] = 3650;
}

}
else
{
	if(input[i]==(3649+offset0))
{
	vR[i] = 3649;
}
else
{
	vR[i] = 3648;
}

}

}

}

}

}

}
else
{
	if(input[i]>(3615+offset0))
{
	if(input[i]>(3631+offset0))
{
	if(input[i]>(3639+offset0))
{
	if(input[i]>(3643+offset0))
{
	if(input[i]>(3645+offset0))
{
	if(input[i]==(3647+offset0))
{
	vR[i] = 3647;
}
else
{
	vR[i] = 3646;
}

}
else
{
	if(input[i]==(3645+offset0))
{
	vR[i] = 3645;
}
else
{
	vR[i] = 3644;
}

}

}
else
{
	if(input[i]>(3641+offset0))
{
	if(input[i]==(3643+offset0))
{
	vR[i] = 3643;
}
else
{
	vR[i] = 3642;
}

}
else
{
	if(input[i]==(3641+offset0))
{
	vR[i] = 3641;
}
else
{
	vR[i] = 3640;
}

}

}

}
else
{
	if(input[i]>(3635+offset0))
{
	if(input[i]>(3637+offset0))
{
	if(input[i]==(3639+offset0))
{
	vR[i] = 3639;
}
else
{
	vR[i] = 3638;
}

}
else
{
	if(input[i]==(3637+offset0))
{
	vR[i] = 3637;
}
else
{
	vR[i] = 3636;
}

}

}
else
{
	if(input[i]>(3633+offset0))
{
	if(input[i]==(3635+offset0))
{
	vR[i] = 3635;
}
else
{
	vR[i] = 3634;
}

}
else
{
	if(input[i]==(3633+offset0))
{
	vR[i] = 3633;
}
else
{
	vR[i] = 3632;
}

}

}

}

}
else
{
	if(input[i]>(3623+offset0))
{
	if(input[i]>(3627+offset0))
{
	if(input[i]>(3629+offset0))
{
	if(input[i]==(3631+offset0))
{
	vR[i] = 3631;
}
else
{
	vR[i] = 3630;
}

}
else
{
	if(input[i]==(3629+offset0))
{
	vR[i] = 3629;
}
else
{
	vR[i] = 3628;
}

}

}
else
{
	if(input[i]>(3625+offset0))
{
	if(input[i]==(3627+offset0))
{
	vR[i] = 3627;
}
else
{
	vR[i] = 3626;
}

}
else
{
	if(input[i]==(3625+offset0))
{
	vR[i] = 3625;
}
else
{
	vR[i] = 3624;
}

}

}

}
else
{
	if(input[i]>(3619+offset0))
{
	if(input[i]>(3621+offset0))
{
	if(input[i]==(3623+offset0))
{
	vR[i] = 3623;
}
else
{
	vR[i] = 3622;
}

}
else
{
	if(input[i]==(3621+offset0))
{
	vR[i] = 3621;
}
else
{
	vR[i] = 3620;
}

}

}
else
{
	if(input[i]>(3617+offset0))
{
	if(input[i]==(3619+offset0))
{
	vR[i] = 3619;
}
else
{
	vR[i] = 3618;
}

}
else
{
	if(input[i]==(3617+offset0))
{
	vR[i] = 3617;
}
else
{
	vR[i] = 3616;
}

}

}

}

}

}
else
{
	if(input[i]>(3599+offset0))
{
	if(input[i]>(3607+offset0))
{
	if(input[i]>(3611+offset0))
{
	if(input[i]>(3613+offset0))
{
	if(input[i]==(3615+offset0))
{
	vR[i] = 3615;
}
else
{
	vR[i] = 3614;
}

}
else
{
	if(input[i]==(3613+offset0))
{
	vR[i] = 3613;
}
else
{
	vR[i] = 3612;
}

}

}
else
{
	if(input[i]>(3609+offset0))
{
	if(input[i]==(3611+offset0))
{
	vR[i] = 3611;
}
else
{
	vR[i] = 3610;
}

}
else
{
	if(input[i]==(3609+offset0))
{
	vR[i] = 3609;
}
else
{
	vR[i] = 3608;
}

}

}

}
else
{
	if(input[i]>(3603+offset0))
{
	if(input[i]>(3605+offset0))
{
	if(input[i]==(3607+offset0))
{
	vR[i] = 3607;
}
else
{
	vR[i] = 3606;
}

}
else
{
	if(input[i]==(3605+offset0))
{
	vR[i] = 3605;
}
else
{
	vR[i] = 3604;
}

}

}
else
{
	if(input[i]>(3601+offset0))
{
	if(input[i]==(3603+offset0))
{
	vR[i] = 3603;
}
else
{
	vR[i] = 3602;
}

}
else
{
	if(input[i]==(3601+offset0))
{
	vR[i] = 3601;
}
else
{
	vR[i] = 3600;
}

}

}

}

}
else
{
	if(input[i]>(3591+offset0))
{
	if(input[i]>(3595+offset0))
{
	if(input[i]>(3597+offset0))
{
	if(input[i]==(3599+offset0))
{
	vR[i] = 3599;
}
else
{
	vR[i] = 3598;
}

}
else
{
	if(input[i]==(3597+offset0))
{
	vR[i] = 3597;
}
else
{
	vR[i] = 3596;
}

}

}
else
{
	if(input[i]>(3593+offset0))
{
	if(input[i]==(3595+offset0))
{
	vR[i] = 3595;
}
else
{
	vR[i] = 3594;
}

}
else
{
	if(input[i]==(3593+offset0))
{
	vR[i] = 3593;
}
else
{
	vR[i] = 3592;
}

}

}

}
else
{
	if(input[i]>(3587+offset0))
{
	if(input[i]>(3589+offset0))
{
	if(input[i]==(3591+offset0))
{
	vR[i] = 3591;
}
else
{
	vR[i] = 3590;
}

}
else
{
	if(input[i]==(3589+offset0))
{
	vR[i] = 3589;
}
else
{
	vR[i] = 3588;
}

}

}
else
{
	if(input[i]>(3585+offset0))
{
	if(input[i]==(3587+offset0))
{
	vR[i] = 3587;
}
else
{
	vR[i] = 3586;
}

}
else
{
	if(input[i]==(3585+offset0))
{
	vR[i] = 3585;
}
else
{
	vR[i] = 3584;
}

}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(3327+offset0))
{
	if(input[i]>(3455+offset0))
{
	if(input[i]>(3519+offset0))
{
	if(input[i]>(3551+offset0))
{
	if(input[i]>(3567+offset0))
{
	if(input[i]>(3575+offset0))
{
	if(input[i]>(3579+offset0))
{
	if(input[i]>(3581+offset0))
{
	if(input[i]==(3583+offset0))
{
	vR[i] = 3583;
}
else
{
	vR[i] = 3582;
}

}
else
{
	if(input[i]==(3581+offset0))
{
	vR[i] = 3581;
}
else
{
	vR[i] = 3580;
}

}

}
else
{
	if(input[i]>(3577+offset0))
{
	if(input[i]==(3579+offset0))
{
	vR[i] = 3579;
}
else
{
	vR[i] = 3578;
}

}
else
{
	if(input[i]==(3577+offset0))
{
	vR[i] = 3577;
}
else
{
	vR[i] = 3576;
}

}

}

}
else
{
	if(input[i]>(3571+offset0))
{
	if(input[i]>(3573+offset0))
{
	if(input[i]==(3575+offset0))
{
	vR[i] = 3575;
}
else
{
	vR[i] = 3574;
}

}
else
{
	if(input[i]==(3573+offset0))
{
	vR[i] = 3573;
}
else
{
	vR[i] = 3572;
}

}

}
else
{
	if(input[i]>(3569+offset0))
{
	if(input[i]==(3571+offset0))
{
	vR[i] = 3571;
}
else
{
	vR[i] = 3570;
}

}
else
{
	if(input[i]==(3569+offset0))
{
	vR[i] = 3569;
}
else
{
	vR[i] = 3568;
}

}

}

}

}
else
{
	if(input[i]>(3559+offset0))
{
	if(input[i]>(3563+offset0))
{
	if(input[i]>(3565+offset0))
{
	if(input[i]==(3567+offset0))
{
	vR[i] = 3567;
}
else
{
	vR[i] = 3566;
}

}
else
{
	if(input[i]==(3565+offset0))
{
	vR[i] = 3565;
}
else
{
	vR[i] = 3564;
}

}

}
else
{
	if(input[i]>(3561+offset0))
{
	if(input[i]==(3563+offset0))
{
	vR[i] = 3563;
}
else
{
	vR[i] = 3562;
}

}
else
{
	if(input[i]==(3561+offset0))
{
	vR[i] = 3561;
}
else
{
	vR[i] = 3560;
}

}

}

}
else
{
	if(input[i]>(3555+offset0))
{
	if(input[i]>(3557+offset0))
{
	if(input[i]==(3559+offset0))
{
	vR[i] = 3559;
}
else
{
	vR[i] = 3558;
}

}
else
{
	if(input[i]==(3557+offset0))
{
	vR[i] = 3557;
}
else
{
	vR[i] = 3556;
}

}

}
else
{
	if(input[i]>(3553+offset0))
{
	if(input[i]==(3555+offset0))
{
	vR[i] = 3555;
}
else
{
	vR[i] = 3554;
}

}
else
{
	if(input[i]==(3553+offset0))
{
	vR[i] = 3553;
}
else
{
	vR[i] = 3552;
}

}

}

}

}

}
else
{
	if(input[i]>(3535+offset0))
{
	if(input[i]>(3543+offset0))
{
	if(input[i]>(3547+offset0))
{
	if(input[i]>(3549+offset0))
{
	if(input[i]==(3551+offset0))
{
	vR[i] = 3551;
}
else
{
	vR[i] = 3550;
}

}
else
{
	if(input[i]==(3549+offset0))
{
	vR[i] = 3549;
}
else
{
	vR[i] = 3548;
}

}

}
else
{
	if(input[i]>(3545+offset0))
{
	if(input[i]==(3547+offset0))
{
	vR[i] = 3547;
}
else
{
	vR[i] = 3546;
}

}
else
{
	if(input[i]==(3545+offset0))
{
	vR[i] = 3545;
}
else
{
	vR[i] = 3544;
}

}

}

}
else
{
	if(input[i]>(3539+offset0))
{
	if(input[i]>(3541+offset0))
{
	if(input[i]==(3543+offset0))
{
	vR[i] = 3543;
}
else
{
	vR[i] = 3542;
}

}
else
{
	if(input[i]==(3541+offset0))
{
	vR[i] = 3541;
}
else
{
	vR[i] = 3540;
}

}

}
else
{
	if(input[i]>(3537+offset0))
{
	if(input[i]==(3539+offset0))
{
	vR[i] = 3539;
}
else
{
	vR[i] = 3538;
}

}
else
{
	if(input[i]==(3537+offset0))
{
	vR[i] = 3537;
}
else
{
	vR[i] = 3536;
}

}

}

}

}
else
{
	if(input[i]>(3527+offset0))
{
	if(input[i]>(3531+offset0))
{
	if(input[i]>(3533+offset0))
{
	if(input[i]==(3535+offset0))
{
	vR[i] = 3535;
}
else
{
	vR[i] = 3534;
}

}
else
{
	if(input[i]==(3533+offset0))
{
	vR[i] = 3533;
}
else
{
	vR[i] = 3532;
}

}

}
else
{
	if(input[i]>(3529+offset0))
{
	if(input[i]==(3531+offset0))
{
	vR[i] = 3531;
}
else
{
	vR[i] = 3530;
}

}
else
{
	if(input[i]==(3529+offset0))
{
	vR[i] = 3529;
}
else
{
	vR[i] = 3528;
}

}

}

}
else
{
	if(input[i]>(3523+offset0))
{
	if(input[i]>(3525+offset0))
{
	if(input[i]==(3527+offset0))
{
	vR[i] = 3527;
}
else
{
	vR[i] = 3526;
}

}
else
{
	if(input[i]==(3525+offset0))
{
	vR[i] = 3525;
}
else
{
	vR[i] = 3524;
}

}

}
else
{
	if(input[i]>(3521+offset0))
{
	if(input[i]==(3523+offset0))
{
	vR[i] = 3523;
}
else
{
	vR[i] = 3522;
}

}
else
{
	if(input[i]==(3521+offset0))
{
	vR[i] = 3521;
}
else
{
	vR[i] = 3520;
}

}

}

}

}

}

}
else
{
	if(input[i]>(3487+offset0))
{
	if(input[i]>(3503+offset0))
{
	if(input[i]>(3511+offset0))
{
	if(input[i]>(3515+offset0))
{
	if(input[i]>(3517+offset0))
{
	if(input[i]==(3519+offset0))
{
	vR[i] = 3519;
}
else
{
	vR[i] = 3518;
}

}
else
{
	if(input[i]==(3517+offset0))
{
	vR[i] = 3517;
}
else
{
	vR[i] = 3516;
}

}

}
else
{
	if(input[i]>(3513+offset0))
{
	if(input[i]==(3515+offset0))
{
	vR[i] = 3515;
}
else
{
	vR[i] = 3514;
}

}
else
{
	if(input[i]==(3513+offset0))
{
	vR[i] = 3513;
}
else
{
	vR[i] = 3512;
}

}

}

}
else
{
	if(input[i]>(3507+offset0))
{
	if(input[i]>(3509+offset0))
{
	if(input[i]==(3511+offset0))
{
	vR[i] = 3511;
}
else
{
	vR[i] = 3510;
}

}
else
{
	if(input[i]==(3509+offset0))
{
	vR[i] = 3509;
}
else
{
	vR[i] = 3508;
}

}

}
else
{
	if(input[i]>(3505+offset0))
{
	if(input[i]==(3507+offset0))
{
	vR[i] = 3507;
}
else
{
	vR[i] = 3506;
}

}
else
{
	if(input[i]==(3505+offset0))
{
	vR[i] = 3505;
}
else
{
	vR[i] = 3504;
}

}

}

}

}
else
{
	if(input[i]>(3495+offset0))
{
	if(input[i]>(3499+offset0))
{
	if(input[i]>(3501+offset0))
{
	if(input[i]==(3503+offset0))
{
	vR[i] = 3503;
}
else
{
	vR[i] = 3502;
}

}
else
{
	if(input[i]==(3501+offset0))
{
	vR[i] = 3501;
}
else
{
	vR[i] = 3500;
}

}

}
else
{
	if(input[i]>(3497+offset0))
{
	if(input[i]==(3499+offset0))
{
	vR[i] = 3499;
}
else
{
	vR[i] = 3498;
}

}
else
{
	if(input[i]==(3497+offset0))
{
	vR[i] = 3497;
}
else
{
	vR[i] = 3496;
}

}

}

}
else
{
	if(input[i]>(3491+offset0))
{
	if(input[i]>(3493+offset0))
{
	if(input[i]==(3495+offset0))
{
	vR[i] = 3495;
}
else
{
	vR[i] = 3494;
}

}
else
{
	if(input[i]==(3493+offset0))
{
	vR[i] = 3493;
}
else
{
	vR[i] = 3492;
}

}

}
else
{
	if(input[i]>(3489+offset0))
{
	if(input[i]==(3491+offset0))
{
	vR[i] = 3491;
}
else
{
	vR[i] = 3490;
}

}
else
{
	if(input[i]==(3489+offset0))
{
	vR[i] = 3489;
}
else
{
	vR[i] = 3488;
}

}

}

}

}

}
else
{
	if(input[i]>(3471+offset0))
{
	if(input[i]>(3479+offset0))
{
	if(input[i]>(3483+offset0))
{
	if(input[i]>(3485+offset0))
{
	if(input[i]==(3487+offset0))
{
	vR[i] = 3487;
}
else
{
	vR[i] = 3486;
}

}
else
{
	if(input[i]==(3485+offset0))
{
	vR[i] = 3485;
}
else
{
	vR[i] = 3484;
}

}

}
else
{
	if(input[i]>(3481+offset0))
{
	if(input[i]==(3483+offset0))
{
	vR[i] = 3483;
}
else
{
	vR[i] = 3482;
}

}
else
{
	if(input[i]==(3481+offset0))
{
	vR[i] = 3481;
}
else
{
	vR[i] = 3480;
}

}

}

}
else
{
	if(input[i]>(3475+offset0))
{
	if(input[i]>(3477+offset0))
{
	if(input[i]==(3479+offset0))
{
	vR[i] = 3479;
}
else
{
	vR[i] = 3478;
}

}
else
{
	if(input[i]==(3477+offset0))
{
	vR[i] = 3477;
}
else
{
	vR[i] = 3476;
}

}

}
else
{
	if(input[i]>(3473+offset0))
{
	if(input[i]==(3475+offset0))
{
	vR[i] = 3475;
}
else
{
	vR[i] = 3474;
}

}
else
{
	if(input[i]==(3473+offset0))
{
	vR[i] = 3473;
}
else
{
	vR[i] = 3472;
}

}

}

}

}
else
{
	if(input[i]>(3463+offset0))
{
	if(input[i]>(3467+offset0))
{
	if(input[i]>(3469+offset0))
{
	if(input[i]==(3471+offset0))
{
	vR[i] = 3471;
}
else
{
	vR[i] = 3470;
}

}
else
{
	if(input[i]==(3469+offset0))
{
	vR[i] = 3469;
}
else
{
	vR[i] = 3468;
}

}

}
else
{
	if(input[i]>(3465+offset0))
{
	if(input[i]==(3467+offset0))
{
	vR[i] = 3467;
}
else
{
	vR[i] = 3466;
}

}
else
{
	if(input[i]==(3465+offset0))
{
	vR[i] = 3465;
}
else
{
	vR[i] = 3464;
}

}

}

}
else
{
	if(input[i]>(3459+offset0))
{
	if(input[i]>(3461+offset0))
{
	if(input[i]==(3463+offset0))
{
	vR[i] = 3463;
}
else
{
	vR[i] = 3462;
}

}
else
{
	if(input[i]==(3461+offset0))
{
	vR[i] = 3461;
}
else
{
	vR[i] = 3460;
}

}

}
else
{
	if(input[i]>(3457+offset0))
{
	if(input[i]==(3459+offset0))
{
	vR[i] = 3459;
}
else
{
	vR[i] = 3458;
}

}
else
{
	if(input[i]==(3457+offset0))
{
	vR[i] = 3457;
}
else
{
	vR[i] = 3456;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(3391+offset0))
{
	if(input[i]>(3423+offset0))
{
	if(input[i]>(3439+offset0))
{
	if(input[i]>(3447+offset0))
{
	if(input[i]>(3451+offset0))
{
	if(input[i]>(3453+offset0))
{
	if(input[i]==(3455+offset0))
{
	vR[i] = 3455;
}
else
{
	vR[i] = 3454;
}

}
else
{
	if(input[i]==(3453+offset0))
{
	vR[i] = 3453;
}
else
{
	vR[i] = 3452;
}

}

}
else
{
	if(input[i]>(3449+offset0))
{
	if(input[i]==(3451+offset0))
{
	vR[i] = 3451;
}
else
{
	vR[i] = 3450;
}

}
else
{
	if(input[i]==(3449+offset0))
{
	vR[i] = 3449;
}
else
{
	vR[i] = 3448;
}

}

}

}
else
{
	if(input[i]>(3443+offset0))
{
	if(input[i]>(3445+offset0))
{
	if(input[i]==(3447+offset0))
{
	vR[i] = 3447;
}
else
{
	vR[i] = 3446;
}

}
else
{
	if(input[i]==(3445+offset0))
{
	vR[i] = 3445;
}
else
{
	vR[i] = 3444;
}

}

}
else
{
	if(input[i]>(3441+offset0))
{
	if(input[i]==(3443+offset0))
{
	vR[i] = 3443;
}
else
{
	vR[i] = 3442;
}

}
else
{
	if(input[i]==(3441+offset0))
{
	vR[i] = 3441;
}
else
{
	vR[i] = 3440;
}

}

}

}

}
else
{
	if(input[i]>(3431+offset0))
{
	if(input[i]>(3435+offset0))
{
	if(input[i]>(3437+offset0))
{
	if(input[i]==(3439+offset0))
{
	vR[i] = 3439;
}
else
{
	vR[i] = 3438;
}

}
else
{
	if(input[i]==(3437+offset0))
{
	vR[i] = 3437;
}
else
{
	vR[i] = 3436;
}

}

}
else
{
	if(input[i]>(3433+offset0))
{
	if(input[i]==(3435+offset0))
{
	vR[i] = 3435;
}
else
{
	vR[i] = 3434;
}

}
else
{
	if(input[i]==(3433+offset0))
{
	vR[i] = 3433;
}
else
{
	vR[i] = 3432;
}

}

}

}
else
{
	if(input[i]>(3427+offset0))
{
	if(input[i]>(3429+offset0))
{
	if(input[i]==(3431+offset0))
{
	vR[i] = 3431;
}
else
{
	vR[i] = 3430;
}

}
else
{
	if(input[i]==(3429+offset0))
{
	vR[i] = 3429;
}
else
{
	vR[i] = 3428;
}

}

}
else
{
	if(input[i]>(3425+offset0))
{
	if(input[i]==(3427+offset0))
{
	vR[i] = 3427;
}
else
{
	vR[i] = 3426;
}

}
else
{
	if(input[i]==(3425+offset0))
{
	vR[i] = 3425;
}
else
{
	vR[i] = 3424;
}

}

}

}

}

}
else
{
	if(input[i]>(3407+offset0))
{
	if(input[i]>(3415+offset0))
{
	if(input[i]>(3419+offset0))
{
	if(input[i]>(3421+offset0))
{
	if(input[i]==(3423+offset0))
{
	vR[i] = 3423;
}
else
{
	vR[i] = 3422;
}

}
else
{
	if(input[i]==(3421+offset0))
{
	vR[i] = 3421;
}
else
{
	vR[i] = 3420;
}

}

}
else
{
	if(input[i]>(3417+offset0))
{
	if(input[i]==(3419+offset0))
{
	vR[i] = 3419;
}
else
{
	vR[i] = 3418;
}

}
else
{
	if(input[i]==(3417+offset0))
{
	vR[i] = 3417;
}
else
{
	vR[i] = 3416;
}

}

}

}
else
{
	if(input[i]>(3411+offset0))
{
	if(input[i]>(3413+offset0))
{
	if(input[i]==(3415+offset0))
{
	vR[i] = 3415;
}
else
{
	vR[i] = 3414;
}

}
else
{
	if(input[i]==(3413+offset0))
{
	vR[i] = 3413;
}
else
{
	vR[i] = 3412;
}

}

}
else
{
	if(input[i]>(3409+offset0))
{
	if(input[i]==(3411+offset0))
{
	vR[i] = 3411;
}
else
{
	vR[i] = 3410;
}

}
else
{
	if(input[i]==(3409+offset0))
{
	vR[i] = 3409;
}
else
{
	vR[i] = 3408;
}

}

}

}

}
else
{
	if(input[i]>(3399+offset0))
{
	if(input[i]>(3403+offset0))
{
	if(input[i]>(3405+offset0))
{
	if(input[i]==(3407+offset0))
{
	vR[i] = 3407;
}
else
{
	vR[i] = 3406;
}

}
else
{
	if(input[i]==(3405+offset0))
{
	vR[i] = 3405;
}
else
{
	vR[i] = 3404;
}

}

}
else
{
	if(input[i]>(3401+offset0))
{
	if(input[i]==(3403+offset0))
{
	vR[i] = 3403;
}
else
{
	vR[i] = 3402;
}

}
else
{
	if(input[i]==(3401+offset0))
{
	vR[i] = 3401;
}
else
{
	vR[i] = 3400;
}

}

}

}
else
{
	if(input[i]>(3395+offset0))
{
	if(input[i]>(3397+offset0))
{
	if(input[i]==(3399+offset0))
{
	vR[i] = 3399;
}
else
{
	vR[i] = 3398;
}

}
else
{
	if(input[i]==(3397+offset0))
{
	vR[i] = 3397;
}
else
{
	vR[i] = 3396;
}

}

}
else
{
	if(input[i]>(3393+offset0))
{
	if(input[i]==(3395+offset0))
{
	vR[i] = 3395;
}
else
{
	vR[i] = 3394;
}

}
else
{
	if(input[i]==(3393+offset0))
{
	vR[i] = 3393;
}
else
{
	vR[i] = 3392;
}

}

}

}

}

}

}
else
{
	if(input[i]>(3359+offset0))
{
	if(input[i]>(3375+offset0))
{
	if(input[i]>(3383+offset0))
{
	if(input[i]>(3387+offset0))
{
	if(input[i]>(3389+offset0))
{
	if(input[i]==(3391+offset0))
{
	vR[i] = 3391;
}
else
{
	vR[i] = 3390;
}

}
else
{
	if(input[i]==(3389+offset0))
{
	vR[i] = 3389;
}
else
{
	vR[i] = 3388;
}

}

}
else
{
	if(input[i]>(3385+offset0))
{
	if(input[i]==(3387+offset0))
{
	vR[i] = 3387;
}
else
{
	vR[i] = 3386;
}

}
else
{
	if(input[i]==(3385+offset0))
{
	vR[i] = 3385;
}
else
{
	vR[i] = 3384;
}

}

}

}
else
{
	if(input[i]>(3379+offset0))
{
	if(input[i]>(3381+offset0))
{
	if(input[i]==(3383+offset0))
{
	vR[i] = 3383;
}
else
{
	vR[i] = 3382;
}

}
else
{
	if(input[i]==(3381+offset0))
{
	vR[i] = 3381;
}
else
{
	vR[i] = 3380;
}

}

}
else
{
	if(input[i]>(3377+offset0))
{
	if(input[i]==(3379+offset0))
{
	vR[i] = 3379;
}
else
{
	vR[i] = 3378;
}

}
else
{
	if(input[i]==(3377+offset0))
{
	vR[i] = 3377;
}
else
{
	vR[i] = 3376;
}

}

}

}

}
else
{
	if(input[i]>(3367+offset0))
{
	if(input[i]>(3371+offset0))
{
	if(input[i]>(3373+offset0))
{
	if(input[i]==(3375+offset0))
{
	vR[i] = 3375;
}
else
{
	vR[i] = 3374;
}

}
else
{
	if(input[i]==(3373+offset0))
{
	vR[i] = 3373;
}
else
{
	vR[i] = 3372;
}

}

}
else
{
	if(input[i]>(3369+offset0))
{
	if(input[i]==(3371+offset0))
{
	vR[i] = 3371;
}
else
{
	vR[i] = 3370;
}

}
else
{
	if(input[i]==(3369+offset0))
{
	vR[i] = 3369;
}
else
{
	vR[i] = 3368;
}

}

}

}
else
{
	if(input[i]>(3363+offset0))
{
	if(input[i]>(3365+offset0))
{
	if(input[i]==(3367+offset0))
{
	vR[i] = 3367;
}
else
{
	vR[i] = 3366;
}

}
else
{
	if(input[i]==(3365+offset0))
{
	vR[i] = 3365;
}
else
{
	vR[i] = 3364;
}

}

}
else
{
	if(input[i]>(3361+offset0))
{
	if(input[i]==(3363+offset0))
{
	vR[i] = 3363;
}
else
{
	vR[i] = 3362;
}

}
else
{
	if(input[i]==(3361+offset0))
{
	vR[i] = 3361;
}
else
{
	vR[i] = 3360;
}

}

}

}

}

}
else
{
	if(input[i]>(3343+offset0))
{
	if(input[i]>(3351+offset0))
{
	if(input[i]>(3355+offset0))
{
	if(input[i]>(3357+offset0))
{
	if(input[i]==(3359+offset0))
{
	vR[i] = 3359;
}
else
{
	vR[i] = 3358;
}

}
else
{
	if(input[i]==(3357+offset0))
{
	vR[i] = 3357;
}
else
{
	vR[i] = 3356;
}

}

}
else
{
	if(input[i]>(3353+offset0))
{
	if(input[i]==(3355+offset0))
{
	vR[i] = 3355;
}
else
{
	vR[i] = 3354;
}

}
else
{
	if(input[i]==(3353+offset0))
{
	vR[i] = 3353;
}
else
{
	vR[i] = 3352;
}

}

}

}
else
{
	if(input[i]>(3347+offset0))
{
	if(input[i]>(3349+offset0))
{
	if(input[i]==(3351+offset0))
{
	vR[i] = 3351;
}
else
{
	vR[i] = 3350;
}

}
else
{
	if(input[i]==(3349+offset0))
{
	vR[i] = 3349;
}
else
{
	vR[i] = 3348;
}

}

}
else
{
	if(input[i]>(3345+offset0))
{
	if(input[i]==(3347+offset0))
{
	vR[i] = 3347;
}
else
{
	vR[i] = 3346;
}

}
else
{
	if(input[i]==(3345+offset0))
{
	vR[i] = 3345;
}
else
{
	vR[i] = 3344;
}

}

}

}

}
else
{
	if(input[i]>(3335+offset0))
{
	if(input[i]>(3339+offset0))
{
	if(input[i]>(3341+offset0))
{
	if(input[i]==(3343+offset0))
{
	vR[i] = 3343;
}
else
{
	vR[i] = 3342;
}

}
else
{
	if(input[i]==(3341+offset0))
{
	vR[i] = 3341;
}
else
{
	vR[i] = 3340;
}

}

}
else
{
	if(input[i]>(3337+offset0))
{
	if(input[i]==(3339+offset0))
{
	vR[i] = 3339;
}
else
{
	vR[i] = 3338;
}

}
else
{
	if(input[i]==(3337+offset0))
{
	vR[i] = 3337;
}
else
{
	vR[i] = 3336;
}

}

}

}
else
{
	if(input[i]>(3331+offset0))
{
	if(input[i]>(3333+offset0))
{
	if(input[i]==(3335+offset0))
{
	vR[i] = 3335;
}
else
{
	vR[i] = 3334;
}

}
else
{
	if(input[i]==(3333+offset0))
{
	vR[i] = 3333;
}
else
{
	vR[i] = 3332;
}

}

}
else
{
	if(input[i]>(3329+offset0))
{
	if(input[i]==(3331+offset0))
{
	vR[i] = 3331;
}
else
{
	vR[i] = 3330;
}

}
else
{
	if(input[i]==(3329+offset0))
{
	vR[i] = 3329;
}
else
{
	vR[i] = 3328;
}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(3199+offset0))
{
	if(input[i]>(3263+offset0))
{
	if(input[i]>(3295+offset0))
{
	if(input[i]>(3311+offset0))
{
	if(input[i]>(3319+offset0))
{
	if(input[i]>(3323+offset0))
{
	if(input[i]>(3325+offset0))
{
	if(input[i]==(3327+offset0))
{
	vR[i] = 3327;
}
else
{
	vR[i] = 3326;
}

}
else
{
	if(input[i]==(3325+offset0))
{
	vR[i] = 3325;
}
else
{
	vR[i] = 3324;
}

}

}
else
{
	if(input[i]>(3321+offset0))
{
	if(input[i]==(3323+offset0))
{
	vR[i] = 3323;
}
else
{
	vR[i] = 3322;
}

}
else
{
	if(input[i]==(3321+offset0))
{
	vR[i] = 3321;
}
else
{
	vR[i] = 3320;
}

}

}

}
else
{
	if(input[i]>(3315+offset0))
{
	if(input[i]>(3317+offset0))
{
	if(input[i]==(3319+offset0))
{
	vR[i] = 3319;
}
else
{
	vR[i] = 3318;
}

}
else
{
	if(input[i]==(3317+offset0))
{
	vR[i] = 3317;
}
else
{
	vR[i] = 3316;
}

}

}
else
{
	if(input[i]>(3313+offset0))
{
	if(input[i]==(3315+offset0))
{
	vR[i] = 3315;
}
else
{
	vR[i] = 3314;
}

}
else
{
	if(input[i]==(3313+offset0))
{
	vR[i] = 3313;
}
else
{
	vR[i] = 3312;
}

}

}

}

}
else
{
	if(input[i]>(3303+offset0))
{
	if(input[i]>(3307+offset0))
{
	if(input[i]>(3309+offset0))
{
	if(input[i]==(3311+offset0))
{
	vR[i] = 3311;
}
else
{
	vR[i] = 3310;
}

}
else
{
	if(input[i]==(3309+offset0))
{
	vR[i] = 3309;
}
else
{
	vR[i] = 3308;
}

}

}
else
{
	if(input[i]>(3305+offset0))
{
	if(input[i]==(3307+offset0))
{
	vR[i] = 3307;
}
else
{
	vR[i] = 3306;
}

}
else
{
	if(input[i]==(3305+offset0))
{
	vR[i] = 3305;
}
else
{
	vR[i] = 3304;
}

}

}

}
else
{
	if(input[i]>(3299+offset0))
{
	if(input[i]>(3301+offset0))
{
	if(input[i]==(3303+offset0))
{
	vR[i] = 3303;
}
else
{
	vR[i] = 3302;
}

}
else
{
	if(input[i]==(3301+offset0))
{
	vR[i] = 3301;
}
else
{
	vR[i] = 3300;
}

}

}
else
{
	if(input[i]>(3297+offset0))
{
	if(input[i]==(3299+offset0))
{
	vR[i] = 3299;
}
else
{
	vR[i] = 3298;
}

}
else
{
	if(input[i]==(3297+offset0))
{
	vR[i] = 3297;
}
else
{
	vR[i] = 3296;
}

}

}

}

}

}
else
{
	if(input[i]>(3279+offset0))
{
	if(input[i]>(3287+offset0))
{
	if(input[i]>(3291+offset0))
{
	if(input[i]>(3293+offset0))
{
	if(input[i]==(3295+offset0))
{
	vR[i] = 3295;
}
else
{
	vR[i] = 3294;
}

}
else
{
	if(input[i]==(3293+offset0))
{
	vR[i] = 3293;
}
else
{
	vR[i] = 3292;
}

}

}
else
{
	if(input[i]>(3289+offset0))
{
	if(input[i]==(3291+offset0))
{
	vR[i] = 3291;
}
else
{
	vR[i] = 3290;
}

}
else
{
	if(input[i]==(3289+offset0))
{
	vR[i] = 3289;
}
else
{
	vR[i] = 3288;
}

}

}

}
else
{
	if(input[i]>(3283+offset0))
{
	if(input[i]>(3285+offset0))
{
	if(input[i]==(3287+offset0))
{
	vR[i] = 3287;
}
else
{
	vR[i] = 3286;
}

}
else
{
	if(input[i]==(3285+offset0))
{
	vR[i] = 3285;
}
else
{
	vR[i] = 3284;
}

}

}
else
{
	if(input[i]>(3281+offset0))
{
	if(input[i]==(3283+offset0))
{
	vR[i] = 3283;
}
else
{
	vR[i] = 3282;
}

}
else
{
	if(input[i]==(3281+offset0))
{
	vR[i] = 3281;
}
else
{
	vR[i] = 3280;
}

}

}

}

}
else
{
	if(input[i]>(3271+offset0))
{
	if(input[i]>(3275+offset0))
{
	if(input[i]>(3277+offset0))
{
	if(input[i]==(3279+offset0))
{
	vR[i] = 3279;
}
else
{
	vR[i] = 3278;
}

}
else
{
	if(input[i]==(3277+offset0))
{
	vR[i] = 3277;
}
else
{
	vR[i] = 3276;
}

}

}
else
{
	if(input[i]>(3273+offset0))
{
	if(input[i]==(3275+offset0))
{
	vR[i] = 3275;
}
else
{
	vR[i] = 3274;
}

}
else
{
	if(input[i]==(3273+offset0))
{
	vR[i] = 3273;
}
else
{
	vR[i] = 3272;
}

}

}

}
else
{
	if(input[i]>(3267+offset0))
{
	if(input[i]>(3269+offset0))
{
	if(input[i]==(3271+offset0))
{
	vR[i] = 3271;
}
else
{
	vR[i] = 3270;
}

}
else
{
	if(input[i]==(3269+offset0))
{
	vR[i] = 3269;
}
else
{
	vR[i] = 3268;
}

}

}
else
{
	if(input[i]>(3265+offset0))
{
	if(input[i]==(3267+offset0))
{
	vR[i] = 3267;
}
else
{
	vR[i] = 3266;
}

}
else
{
	if(input[i]==(3265+offset0))
{
	vR[i] = 3265;
}
else
{
	vR[i] = 3264;
}

}

}

}

}

}

}
else
{
	if(input[i]>(3231+offset0))
{
	if(input[i]>(3247+offset0))
{
	if(input[i]>(3255+offset0))
{
	if(input[i]>(3259+offset0))
{
	if(input[i]>(3261+offset0))
{
	if(input[i]==(3263+offset0))
{
	vR[i] = 3263;
}
else
{
	vR[i] = 3262;
}

}
else
{
	if(input[i]==(3261+offset0))
{
	vR[i] = 3261;
}
else
{
	vR[i] = 3260;
}

}

}
else
{
	if(input[i]>(3257+offset0))
{
	if(input[i]==(3259+offset0))
{
	vR[i] = 3259;
}
else
{
	vR[i] = 3258;
}

}
else
{
	if(input[i]==(3257+offset0))
{
	vR[i] = 3257;
}
else
{
	vR[i] = 3256;
}

}

}

}
else
{
	if(input[i]>(3251+offset0))
{
	if(input[i]>(3253+offset0))
{
	if(input[i]==(3255+offset0))
{
	vR[i] = 3255;
}
else
{
	vR[i] = 3254;
}

}
else
{
	if(input[i]==(3253+offset0))
{
	vR[i] = 3253;
}
else
{
	vR[i] = 3252;
}

}

}
else
{
	if(input[i]>(3249+offset0))
{
	if(input[i]==(3251+offset0))
{
	vR[i] = 3251;
}
else
{
	vR[i] = 3250;
}

}
else
{
	if(input[i]==(3249+offset0))
{
	vR[i] = 3249;
}
else
{
	vR[i] = 3248;
}

}

}

}

}
else
{
	if(input[i]>(3239+offset0))
{
	if(input[i]>(3243+offset0))
{
	if(input[i]>(3245+offset0))
{
	if(input[i]==(3247+offset0))
{
	vR[i] = 3247;
}
else
{
	vR[i] = 3246;
}

}
else
{
	if(input[i]==(3245+offset0))
{
	vR[i] = 3245;
}
else
{
	vR[i] = 3244;
}

}

}
else
{
	if(input[i]>(3241+offset0))
{
	if(input[i]==(3243+offset0))
{
	vR[i] = 3243;
}
else
{
	vR[i] = 3242;
}

}
else
{
	if(input[i]==(3241+offset0))
{
	vR[i] = 3241;
}
else
{
	vR[i] = 3240;
}

}

}

}
else
{
	if(input[i]>(3235+offset0))
{
	if(input[i]>(3237+offset0))
{
	if(input[i]==(3239+offset0))
{
	vR[i] = 3239;
}
else
{
	vR[i] = 3238;
}

}
else
{
	if(input[i]==(3237+offset0))
{
	vR[i] = 3237;
}
else
{
	vR[i] = 3236;
}

}

}
else
{
	if(input[i]>(3233+offset0))
{
	if(input[i]==(3235+offset0))
{
	vR[i] = 3235;
}
else
{
	vR[i] = 3234;
}

}
else
{
	if(input[i]==(3233+offset0))
{
	vR[i] = 3233;
}
else
{
	vR[i] = 3232;
}

}

}

}

}

}
else
{
	if(input[i]>(3215+offset0))
{
	if(input[i]>(3223+offset0))
{
	if(input[i]>(3227+offset0))
{
	if(input[i]>(3229+offset0))
{
	if(input[i]==(3231+offset0))
{
	vR[i] = 3231;
}
else
{
	vR[i] = 3230;
}

}
else
{
	if(input[i]==(3229+offset0))
{
	vR[i] = 3229;
}
else
{
	vR[i] = 3228;
}

}

}
else
{
	if(input[i]>(3225+offset0))
{
	if(input[i]==(3227+offset0))
{
	vR[i] = 3227;
}
else
{
	vR[i] = 3226;
}

}
else
{
	if(input[i]==(3225+offset0))
{
	vR[i] = 3225;
}
else
{
	vR[i] = 3224;
}

}

}

}
else
{
	if(input[i]>(3219+offset0))
{
	if(input[i]>(3221+offset0))
{
	if(input[i]==(3223+offset0))
{
	vR[i] = 3223;
}
else
{
	vR[i] = 3222;
}

}
else
{
	if(input[i]==(3221+offset0))
{
	vR[i] = 3221;
}
else
{
	vR[i] = 3220;
}

}

}
else
{
	if(input[i]>(3217+offset0))
{
	if(input[i]==(3219+offset0))
{
	vR[i] = 3219;
}
else
{
	vR[i] = 3218;
}

}
else
{
	if(input[i]==(3217+offset0))
{
	vR[i] = 3217;
}
else
{
	vR[i] = 3216;
}

}

}

}

}
else
{
	if(input[i]>(3207+offset0))
{
	if(input[i]>(3211+offset0))
{
	if(input[i]>(3213+offset0))
{
	if(input[i]==(3215+offset0))
{
	vR[i] = 3215;
}
else
{
	vR[i] = 3214;
}

}
else
{
	if(input[i]==(3213+offset0))
{
	vR[i] = 3213;
}
else
{
	vR[i] = 3212;
}

}

}
else
{
	if(input[i]>(3209+offset0))
{
	if(input[i]==(3211+offset0))
{
	vR[i] = 3211;
}
else
{
	vR[i] = 3210;
}

}
else
{
	if(input[i]==(3209+offset0))
{
	vR[i] = 3209;
}
else
{
	vR[i] = 3208;
}

}

}

}
else
{
	if(input[i]>(3203+offset0))
{
	if(input[i]>(3205+offset0))
{
	if(input[i]==(3207+offset0))
{
	vR[i] = 3207;
}
else
{
	vR[i] = 3206;
}

}
else
{
	if(input[i]==(3205+offset0))
{
	vR[i] = 3205;
}
else
{
	vR[i] = 3204;
}

}

}
else
{
	if(input[i]>(3201+offset0))
{
	if(input[i]==(3203+offset0))
{
	vR[i] = 3203;
}
else
{
	vR[i] = 3202;
}

}
else
{
	if(input[i]==(3201+offset0))
{
	vR[i] = 3201;
}
else
{
	vR[i] = 3200;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(3135+offset0))
{
	if(input[i]>(3167+offset0))
{
	if(input[i]>(3183+offset0))
{
	if(input[i]>(3191+offset0))
{
	if(input[i]>(3195+offset0))
{
	if(input[i]>(3197+offset0))
{
	if(input[i]==(3199+offset0))
{
	vR[i] = 3199;
}
else
{
	vR[i] = 3198;
}

}
else
{
	if(input[i]==(3197+offset0))
{
	vR[i] = 3197;
}
else
{
	vR[i] = 3196;
}

}

}
else
{
	if(input[i]>(3193+offset0))
{
	if(input[i]==(3195+offset0))
{
	vR[i] = 3195;
}
else
{
	vR[i] = 3194;
}

}
else
{
	if(input[i]==(3193+offset0))
{
	vR[i] = 3193;
}
else
{
	vR[i] = 3192;
}

}

}

}
else
{
	if(input[i]>(3187+offset0))
{
	if(input[i]>(3189+offset0))
{
	if(input[i]==(3191+offset0))
{
	vR[i] = 3191;
}
else
{
	vR[i] = 3190;
}

}
else
{
	if(input[i]==(3189+offset0))
{
	vR[i] = 3189;
}
else
{
	vR[i] = 3188;
}

}

}
else
{
	if(input[i]>(3185+offset0))
{
	if(input[i]==(3187+offset0))
{
	vR[i] = 3187;
}
else
{
	vR[i] = 3186;
}

}
else
{
	if(input[i]==(3185+offset0))
{
	vR[i] = 3185;
}
else
{
	vR[i] = 3184;
}

}

}

}

}
else
{
	if(input[i]>(3175+offset0))
{
	if(input[i]>(3179+offset0))
{
	if(input[i]>(3181+offset0))
{
	if(input[i]==(3183+offset0))
{
	vR[i] = 3183;
}
else
{
	vR[i] = 3182;
}

}
else
{
	if(input[i]==(3181+offset0))
{
	vR[i] = 3181;
}
else
{
	vR[i] = 3180;
}

}

}
else
{
	if(input[i]>(3177+offset0))
{
	if(input[i]==(3179+offset0))
{
	vR[i] = 3179;
}
else
{
	vR[i] = 3178;
}

}
else
{
	if(input[i]==(3177+offset0))
{
	vR[i] = 3177;
}
else
{
	vR[i] = 3176;
}

}

}

}
else
{
	if(input[i]>(3171+offset0))
{
	if(input[i]>(3173+offset0))
{
	if(input[i]==(3175+offset0))
{
	vR[i] = 3175;
}
else
{
	vR[i] = 3174;
}

}
else
{
	if(input[i]==(3173+offset0))
{
	vR[i] = 3173;
}
else
{
	vR[i] = 3172;
}

}

}
else
{
	if(input[i]>(3169+offset0))
{
	if(input[i]==(3171+offset0))
{
	vR[i] = 3171;
}
else
{
	vR[i] = 3170;
}

}
else
{
	if(input[i]==(3169+offset0))
{
	vR[i] = 3169;
}
else
{
	vR[i] = 3168;
}

}

}

}

}

}
else
{
	if(input[i]>(3151+offset0))
{
	if(input[i]>(3159+offset0))
{
	if(input[i]>(3163+offset0))
{
	if(input[i]>(3165+offset0))
{
	if(input[i]==(3167+offset0))
{
	vR[i] = 3167;
}
else
{
	vR[i] = 3166;
}

}
else
{
	if(input[i]==(3165+offset0))
{
	vR[i] = 3165;
}
else
{
	vR[i] = 3164;
}

}

}
else
{
	if(input[i]>(3161+offset0))
{
	if(input[i]==(3163+offset0))
{
	vR[i] = 3163;
}
else
{
	vR[i] = 3162;
}

}
else
{
	if(input[i]==(3161+offset0))
{
	vR[i] = 3161;
}
else
{
	vR[i] = 3160;
}

}

}

}
else
{
	if(input[i]>(3155+offset0))
{
	if(input[i]>(3157+offset0))
{
	if(input[i]==(3159+offset0))
{
	vR[i] = 3159;
}
else
{
	vR[i] = 3158;
}

}
else
{
	if(input[i]==(3157+offset0))
{
	vR[i] = 3157;
}
else
{
	vR[i] = 3156;
}

}

}
else
{
	if(input[i]>(3153+offset0))
{
	if(input[i]==(3155+offset0))
{
	vR[i] = 3155;
}
else
{
	vR[i] = 3154;
}

}
else
{
	if(input[i]==(3153+offset0))
{
	vR[i] = 3153;
}
else
{
	vR[i] = 3152;
}

}

}

}

}
else
{
	if(input[i]>(3143+offset0))
{
	if(input[i]>(3147+offset0))
{
	if(input[i]>(3149+offset0))
{
	if(input[i]==(3151+offset0))
{
	vR[i] = 3151;
}
else
{
	vR[i] = 3150;
}

}
else
{
	if(input[i]==(3149+offset0))
{
	vR[i] = 3149;
}
else
{
	vR[i] = 3148;
}

}

}
else
{
	if(input[i]>(3145+offset0))
{
	if(input[i]==(3147+offset0))
{
	vR[i] = 3147;
}
else
{
	vR[i] = 3146;
}

}
else
{
	if(input[i]==(3145+offset0))
{
	vR[i] = 3145;
}
else
{
	vR[i] = 3144;
}

}

}

}
else
{
	if(input[i]>(3139+offset0))
{
	if(input[i]>(3141+offset0))
{
	if(input[i]==(3143+offset0))
{
	vR[i] = 3143;
}
else
{
	vR[i] = 3142;
}

}
else
{
	if(input[i]==(3141+offset0))
{
	vR[i] = 3141;
}
else
{
	vR[i] = 3140;
}

}

}
else
{
	if(input[i]>(3137+offset0))
{
	if(input[i]==(3139+offset0))
{
	vR[i] = 3139;
}
else
{
	vR[i] = 3138;
}

}
else
{
	if(input[i]==(3137+offset0))
{
	vR[i] = 3137;
}
else
{
	vR[i] = 3136;
}

}

}

}

}

}

}
else
{
	if(input[i]>(3103+offset0))
{
	if(input[i]>(3119+offset0))
{
	if(input[i]>(3127+offset0))
{
	if(input[i]>(3131+offset0))
{
	if(input[i]>(3133+offset0))
{
	if(input[i]==(3135+offset0))
{
	vR[i] = 3135;
}
else
{
	vR[i] = 3134;
}

}
else
{
	if(input[i]==(3133+offset0))
{
	vR[i] = 3133;
}
else
{
	vR[i] = 3132;
}

}

}
else
{
	if(input[i]>(3129+offset0))
{
	if(input[i]==(3131+offset0))
{
	vR[i] = 3131;
}
else
{
	vR[i] = 3130;
}

}
else
{
	if(input[i]==(3129+offset0))
{
	vR[i] = 3129;
}
else
{
	vR[i] = 3128;
}

}

}

}
else
{
	if(input[i]>(3123+offset0))
{
	if(input[i]>(3125+offset0))
{
	if(input[i]==(3127+offset0))
{
	vR[i] = 3127;
}
else
{
	vR[i] = 3126;
}

}
else
{
	if(input[i]==(3125+offset0))
{
	vR[i] = 3125;
}
else
{
	vR[i] = 3124;
}

}

}
else
{
	if(input[i]>(3121+offset0))
{
	if(input[i]==(3123+offset0))
{
	vR[i] = 3123;
}
else
{
	vR[i] = 3122;
}

}
else
{
	if(input[i]==(3121+offset0))
{
	vR[i] = 3121;
}
else
{
	vR[i] = 3120;
}

}

}

}

}
else
{
	if(input[i]>(3111+offset0))
{
	if(input[i]>(3115+offset0))
{
	if(input[i]>(3117+offset0))
{
	if(input[i]==(3119+offset0))
{
	vR[i] = 3119;
}
else
{
	vR[i] = 3118;
}

}
else
{
	if(input[i]==(3117+offset0))
{
	vR[i] = 3117;
}
else
{
	vR[i] = 3116;
}

}

}
else
{
	if(input[i]>(3113+offset0))
{
	if(input[i]==(3115+offset0))
{
	vR[i] = 3115;
}
else
{
	vR[i] = 3114;
}

}
else
{
	if(input[i]==(3113+offset0))
{
	vR[i] = 3113;
}
else
{
	vR[i] = 3112;
}

}

}

}
else
{
	if(input[i]>(3107+offset0))
{
	if(input[i]>(3109+offset0))
{
	if(input[i]==(3111+offset0))
{
	vR[i] = 3111;
}
else
{
	vR[i] = 3110;
}

}
else
{
	if(input[i]==(3109+offset0))
{
	vR[i] = 3109;
}
else
{
	vR[i] = 3108;
}

}

}
else
{
	if(input[i]>(3105+offset0))
{
	if(input[i]==(3107+offset0))
{
	vR[i] = 3107;
}
else
{
	vR[i] = 3106;
}

}
else
{
	if(input[i]==(3105+offset0))
{
	vR[i] = 3105;
}
else
{
	vR[i] = 3104;
}

}

}

}

}

}
else
{
	if(input[i]>(3087+offset0))
{
	if(input[i]>(3095+offset0))
{
	if(input[i]>(3099+offset0))
{
	if(input[i]>(3101+offset0))
{
	if(input[i]==(3103+offset0))
{
	vR[i] = 3103;
}
else
{
	vR[i] = 3102;
}

}
else
{
	if(input[i]==(3101+offset0))
{
	vR[i] = 3101;
}
else
{
	vR[i] = 3100;
}

}

}
else
{
	if(input[i]>(3097+offset0))
{
	if(input[i]==(3099+offset0))
{
	vR[i] = 3099;
}
else
{
	vR[i] = 3098;
}

}
else
{
	if(input[i]==(3097+offset0))
{
	vR[i] = 3097;
}
else
{
	vR[i] = 3096;
}

}

}

}
else
{
	if(input[i]>(3091+offset0))
{
	if(input[i]>(3093+offset0))
{
	if(input[i]==(3095+offset0))
{
	vR[i] = 3095;
}
else
{
	vR[i] = 3094;
}

}
else
{
	if(input[i]==(3093+offset0))
{
	vR[i] = 3093;
}
else
{
	vR[i] = 3092;
}

}

}
else
{
	if(input[i]>(3089+offset0))
{
	if(input[i]==(3091+offset0))
{
	vR[i] = 3091;
}
else
{
	vR[i] = 3090;
}

}
else
{
	if(input[i]==(3089+offset0))
{
	vR[i] = 3089;
}
else
{
	vR[i] = 3088;
}

}

}

}

}
else
{
	if(input[i]>(3079+offset0))
{
	if(input[i]>(3083+offset0))
{
	if(input[i]>(3085+offset0))
{
	if(input[i]==(3087+offset0))
{
	vR[i] = 3087;
}
else
{
	vR[i] = 3086;
}

}
else
{
	if(input[i]==(3085+offset0))
{
	vR[i] = 3085;
}
else
{
	vR[i] = 3084;
}

}

}
else
{
	if(input[i]>(3081+offset0))
{
	if(input[i]==(3083+offset0))
{
	vR[i] = 3083;
}
else
{
	vR[i] = 3082;
}

}
else
{
	if(input[i]==(3081+offset0))
{
	vR[i] = 3081;
}
else
{
	vR[i] = 3080;
}

}

}

}
else
{
	if(input[i]>(3075+offset0))
{
	if(input[i]>(3077+offset0))
{
	if(input[i]==(3079+offset0))
{
	vR[i] = 3079;
}
else
{
	vR[i] = 3078;
}

}
else
{
	if(input[i]==(3077+offset0))
{
	vR[i] = 3077;
}
else
{
	vR[i] = 3076;
}

}

}
else
{
	if(input[i]>(3073+offset0))
{
	if(input[i]==(3075+offset0))
{
	vR[i] = 3075;
}
else
{
	vR[i] = 3074;
}

}
else
{
	if(input[i]==(3073+offset0))
{
	vR[i] = 3073;
}
else
{
	vR[i] = 3072;
}

}

}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(2559+offset0))
{
	if(input[i]>(2815+offset0))
{
	if(input[i]>(2943+offset0))
{
	if(input[i]>(3007+offset0))
{
	if(input[i]>(3039+offset0))
{
	if(input[i]>(3055+offset0))
{
	if(input[i]>(3063+offset0))
{
	if(input[i]>(3067+offset0))
{
	if(input[i]>(3069+offset0))
{
	if(input[i]==(3071+offset0))
{
	vR[i] = 3071;
}
else
{
	vR[i] = 3070;
}

}
else
{
	if(input[i]==(3069+offset0))
{
	vR[i] = 3069;
}
else
{
	vR[i] = 3068;
}

}

}
else
{
	if(input[i]>(3065+offset0))
{
	if(input[i]==(3067+offset0))
{
	vR[i] = 3067;
}
else
{
	vR[i] = 3066;
}

}
else
{
	if(input[i]==(3065+offset0))
{
	vR[i] = 3065;
}
else
{
	vR[i] = 3064;
}

}

}

}
else
{
	if(input[i]>(3059+offset0))
{
	if(input[i]>(3061+offset0))
{
	if(input[i]==(3063+offset0))
{
	vR[i] = 3063;
}
else
{
	vR[i] = 3062;
}

}
else
{
	if(input[i]==(3061+offset0))
{
	vR[i] = 3061;
}
else
{
	vR[i] = 3060;
}

}

}
else
{
	if(input[i]>(3057+offset0))
{
	if(input[i]==(3059+offset0))
{
	vR[i] = 3059;
}
else
{
	vR[i] = 3058;
}

}
else
{
	if(input[i]==(3057+offset0))
{
	vR[i] = 3057;
}
else
{
	vR[i] = 3056;
}

}

}

}

}
else
{
	if(input[i]>(3047+offset0))
{
	if(input[i]>(3051+offset0))
{
	if(input[i]>(3053+offset0))
{
	if(input[i]==(3055+offset0))
{
	vR[i] = 3055;
}
else
{
	vR[i] = 3054;
}

}
else
{
	if(input[i]==(3053+offset0))
{
	vR[i] = 3053;
}
else
{
	vR[i] = 3052;
}

}

}
else
{
	if(input[i]>(3049+offset0))
{
	if(input[i]==(3051+offset0))
{
	vR[i] = 3051;
}
else
{
	vR[i] = 3050;
}

}
else
{
	if(input[i]==(3049+offset0))
{
	vR[i] = 3049;
}
else
{
	vR[i] = 3048;
}

}

}

}
else
{
	if(input[i]>(3043+offset0))
{
	if(input[i]>(3045+offset0))
{
	if(input[i]==(3047+offset0))
{
	vR[i] = 3047;
}
else
{
	vR[i] = 3046;
}

}
else
{
	if(input[i]==(3045+offset0))
{
	vR[i] = 3045;
}
else
{
	vR[i] = 3044;
}

}

}
else
{
	if(input[i]>(3041+offset0))
{
	if(input[i]==(3043+offset0))
{
	vR[i] = 3043;
}
else
{
	vR[i] = 3042;
}

}
else
{
	if(input[i]==(3041+offset0))
{
	vR[i] = 3041;
}
else
{
	vR[i] = 3040;
}

}

}

}

}

}
else
{
	if(input[i]>(3023+offset0))
{
	if(input[i]>(3031+offset0))
{
	if(input[i]>(3035+offset0))
{
	if(input[i]>(3037+offset0))
{
	if(input[i]==(3039+offset0))
{
	vR[i] = 3039;
}
else
{
	vR[i] = 3038;
}

}
else
{
	if(input[i]==(3037+offset0))
{
	vR[i] = 3037;
}
else
{
	vR[i] = 3036;
}

}

}
else
{
	if(input[i]>(3033+offset0))
{
	if(input[i]==(3035+offset0))
{
	vR[i] = 3035;
}
else
{
	vR[i] = 3034;
}

}
else
{
	if(input[i]==(3033+offset0))
{
	vR[i] = 3033;
}
else
{
	vR[i] = 3032;
}

}

}

}
else
{
	if(input[i]>(3027+offset0))
{
	if(input[i]>(3029+offset0))
{
	if(input[i]==(3031+offset0))
{
	vR[i] = 3031;
}
else
{
	vR[i] = 3030;
}

}
else
{
	if(input[i]==(3029+offset0))
{
	vR[i] = 3029;
}
else
{
	vR[i] = 3028;
}

}

}
else
{
	if(input[i]>(3025+offset0))
{
	if(input[i]==(3027+offset0))
{
	vR[i] = 3027;
}
else
{
	vR[i] = 3026;
}

}
else
{
	if(input[i]==(3025+offset0))
{
	vR[i] = 3025;
}
else
{
	vR[i] = 3024;
}

}

}

}

}
else
{
	if(input[i]>(3015+offset0))
{
	if(input[i]>(3019+offset0))
{
	if(input[i]>(3021+offset0))
{
	if(input[i]==(3023+offset0))
{
	vR[i] = 3023;
}
else
{
	vR[i] = 3022;
}

}
else
{
	if(input[i]==(3021+offset0))
{
	vR[i] = 3021;
}
else
{
	vR[i] = 3020;
}

}

}
else
{
	if(input[i]>(3017+offset0))
{
	if(input[i]==(3019+offset0))
{
	vR[i] = 3019;
}
else
{
	vR[i] = 3018;
}

}
else
{
	if(input[i]==(3017+offset0))
{
	vR[i] = 3017;
}
else
{
	vR[i] = 3016;
}

}

}

}
else
{
	if(input[i]>(3011+offset0))
{
	if(input[i]>(3013+offset0))
{
	if(input[i]==(3015+offset0))
{
	vR[i] = 3015;
}
else
{
	vR[i] = 3014;
}

}
else
{
	if(input[i]==(3013+offset0))
{
	vR[i] = 3013;
}
else
{
	vR[i] = 3012;
}

}

}
else
{
	if(input[i]>(3009+offset0))
{
	if(input[i]==(3011+offset0))
{
	vR[i] = 3011;
}
else
{
	vR[i] = 3010;
}

}
else
{
	if(input[i]==(3009+offset0))
{
	vR[i] = 3009;
}
else
{
	vR[i] = 3008;
}

}

}

}

}

}

}
else
{
	if(input[i]>(2975+offset0))
{
	if(input[i]>(2991+offset0))
{
	if(input[i]>(2999+offset0))
{
	if(input[i]>(3003+offset0))
{
	if(input[i]>(3005+offset0))
{
	if(input[i]==(3007+offset0))
{
	vR[i] = 3007;
}
else
{
	vR[i] = 3006;
}

}
else
{
	if(input[i]==(3005+offset0))
{
	vR[i] = 3005;
}
else
{
	vR[i] = 3004;
}

}

}
else
{
	if(input[i]>(3001+offset0))
{
	if(input[i]==(3003+offset0))
{
	vR[i] = 3003;
}
else
{
	vR[i] = 3002;
}

}
else
{
	if(input[i]==(3001+offset0))
{
	vR[i] = 3001;
}
else
{
	vR[i] = 3000;
}

}

}

}
else
{
	if(input[i]>(2995+offset0))
{
	if(input[i]>(2997+offset0))
{
	if(input[i]==(2999+offset0))
{
	vR[i] = 2999;
}
else
{
	vR[i] = 2998;
}

}
else
{
	if(input[i]==(2997+offset0))
{
	vR[i] = 2997;
}
else
{
	vR[i] = 2996;
}

}

}
else
{
	if(input[i]>(2993+offset0))
{
	if(input[i]==(2995+offset0))
{
	vR[i] = 2995;
}
else
{
	vR[i] = 2994;
}

}
else
{
	if(input[i]==(2993+offset0))
{
	vR[i] = 2993;
}
else
{
	vR[i] = 2992;
}

}

}

}

}
else
{
	if(input[i]>(2983+offset0))
{
	if(input[i]>(2987+offset0))
{
	if(input[i]>(2989+offset0))
{
	if(input[i]==(2991+offset0))
{
	vR[i] = 2991;
}
else
{
	vR[i] = 2990;
}

}
else
{
	if(input[i]==(2989+offset0))
{
	vR[i] = 2989;
}
else
{
	vR[i] = 2988;
}

}

}
else
{
	if(input[i]>(2985+offset0))
{
	if(input[i]==(2987+offset0))
{
	vR[i] = 2987;
}
else
{
	vR[i] = 2986;
}

}
else
{
	if(input[i]==(2985+offset0))
{
	vR[i] = 2985;
}
else
{
	vR[i] = 2984;
}

}

}

}
else
{
	if(input[i]>(2979+offset0))
{
	if(input[i]>(2981+offset0))
{
	if(input[i]==(2983+offset0))
{
	vR[i] = 2983;
}
else
{
	vR[i] = 2982;
}

}
else
{
	if(input[i]==(2981+offset0))
{
	vR[i] = 2981;
}
else
{
	vR[i] = 2980;
}

}

}
else
{
	if(input[i]>(2977+offset0))
{
	if(input[i]==(2979+offset0))
{
	vR[i] = 2979;
}
else
{
	vR[i] = 2978;
}

}
else
{
	if(input[i]==(2977+offset0))
{
	vR[i] = 2977;
}
else
{
	vR[i] = 2976;
}

}

}

}

}

}
else
{
	if(input[i]>(2959+offset0))
{
	if(input[i]>(2967+offset0))
{
	if(input[i]>(2971+offset0))
{
	if(input[i]>(2973+offset0))
{
	if(input[i]==(2975+offset0))
{
	vR[i] = 2975;
}
else
{
	vR[i] = 2974;
}

}
else
{
	if(input[i]==(2973+offset0))
{
	vR[i] = 2973;
}
else
{
	vR[i] = 2972;
}

}

}
else
{
	if(input[i]>(2969+offset0))
{
	if(input[i]==(2971+offset0))
{
	vR[i] = 2971;
}
else
{
	vR[i] = 2970;
}

}
else
{
	if(input[i]==(2969+offset0))
{
	vR[i] = 2969;
}
else
{
	vR[i] = 2968;
}

}

}

}
else
{
	if(input[i]>(2963+offset0))
{
	if(input[i]>(2965+offset0))
{
	if(input[i]==(2967+offset0))
{
	vR[i] = 2967;
}
else
{
	vR[i] = 2966;
}

}
else
{
	if(input[i]==(2965+offset0))
{
	vR[i] = 2965;
}
else
{
	vR[i] = 2964;
}

}

}
else
{
	if(input[i]>(2961+offset0))
{
	if(input[i]==(2963+offset0))
{
	vR[i] = 2963;
}
else
{
	vR[i] = 2962;
}

}
else
{
	if(input[i]==(2961+offset0))
{
	vR[i] = 2961;
}
else
{
	vR[i] = 2960;
}

}

}

}

}
else
{
	if(input[i]>(2951+offset0))
{
	if(input[i]>(2955+offset0))
{
	if(input[i]>(2957+offset0))
{
	if(input[i]==(2959+offset0))
{
	vR[i] = 2959;
}
else
{
	vR[i] = 2958;
}

}
else
{
	if(input[i]==(2957+offset0))
{
	vR[i] = 2957;
}
else
{
	vR[i] = 2956;
}

}

}
else
{
	if(input[i]>(2953+offset0))
{
	if(input[i]==(2955+offset0))
{
	vR[i] = 2955;
}
else
{
	vR[i] = 2954;
}

}
else
{
	if(input[i]==(2953+offset0))
{
	vR[i] = 2953;
}
else
{
	vR[i] = 2952;
}

}

}

}
else
{
	if(input[i]>(2947+offset0))
{
	if(input[i]>(2949+offset0))
{
	if(input[i]==(2951+offset0))
{
	vR[i] = 2951;
}
else
{
	vR[i] = 2950;
}

}
else
{
	if(input[i]==(2949+offset0))
{
	vR[i] = 2949;
}
else
{
	vR[i] = 2948;
}

}

}
else
{
	if(input[i]>(2945+offset0))
{
	if(input[i]==(2947+offset0))
{
	vR[i] = 2947;
}
else
{
	vR[i] = 2946;
}

}
else
{
	if(input[i]==(2945+offset0))
{
	vR[i] = 2945;
}
else
{
	vR[i] = 2944;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(2879+offset0))
{
	if(input[i]>(2911+offset0))
{
	if(input[i]>(2927+offset0))
{
	if(input[i]>(2935+offset0))
{
	if(input[i]>(2939+offset0))
{
	if(input[i]>(2941+offset0))
{
	if(input[i]==(2943+offset0))
{
	vR[i] = 2943;
}
else
{
	vR[i] = 2942;
}

}
else
{
	if(input[i]==(2941+offset0))
{
	vR[i] = 2941;
}
else
{
	vR[i] = 2940;
}

}

}
else
{
	if(input[i]>(2937+offset0))
{
	if(input[i]==(2939+offset0))
{
	vR[i] = 2939;
}
else
{
	vR[i] = 2938;
}

}
else
{
	if(input[i]==(2937+offset0))
{
	vR[i] = 2937;
}
else
{
	vR[i] = 2936;
}

}

}

}
else
{
	if(input[i]>(2931+offset0))
{
	if(input[i]>(2933+offset0))
{
	if(input[i]==(2935+offset0))
{
	vR[i] = 2935;
}
else
{
	vR[i] = 2934;
}

}
else
{
	if(input[i]==(2933+offset0))
{
	vR[i] = 2933;
}
else
{
	vR[i] = 2932;
}

}

}
else
{
	if(input[i]>(2929+offset0))
{
	if(input[i]==(2931+offset0))
{
	vR[i] = 2931;
}
else
{
	vR[i] = 2930;
}

}
else
{
	if(input[i]==(2929+offset0))
{
	vR[i] = 2929;
}
else
{
	vR[i] = 2928;
}

}

}

}

}
else
{
	if(input[i]>(2919+offset0))
{
	if(input[i]>(2923+offset0))
{
	if(input[i]>(2925+offset0))
{
	if(input[i]==(2927+offset0))
{
	vR[i] = 2927;
}
else
{
	vR[i] = 2926;
}

}
else
{
	if(input[i]==(2925+offset0))
{
	vR[i] = 2925;
}
else
{
	vR[i] = 2924;
}

}

}
else
{
	if(input[i]>(2921+offset0))
{
	if(input[i]==(2923+offset0))
{
	vR[i] = 2923;
}
else
{
	vR[i] = 2922;
}

}
else
{
	if(input[i]==(2921+offset0))
{
	vR[i] = 2921;
}
else
{
	vR[i] = 2920;
}

}

}

}
else
{
	if(input[i]>(2915+offset0))
{
	if(input[i]>(2917+offset0))
{
	if(input[i]==(2919+offset0))
{
	vR[i] = 2919;
}
else
{
	vR[i] = 2918;
}

}
else
{
	if(input[i]==(2917+offset0))
{
	vR[i] = 2917;
}
else
{
	vR[i] = 2916;
}

}

}
else
{
	if(input[i]>(2913+offset0))
{
	if(input[i]==(2915+offset0))
{
	vR[i] = 2915;
}
else
{
	vR[i] = 2914;
}

}
else
{
	if(input[i]==(2913+offset0))
{
	vR[i] = 2913;
}
else
{
	vR[i] = 2912;
}

}

}

}

}

}
else
{
	if(input[i]>(2895+offset0))
{
	if(input[i]>(2903+offset0))
{
	if(input[i]>(2907+offset0))
{
	if(input[i]>(2909+offset0))
{
	if(input[i]==(2911+offset0))
{
	vR[i] = 2911;
}
else
{
	vR[i] = 2910;
}

}
else
{
	if(input[i]==(2909+offset0))
{
	vR[i] = 2909;
}
else
{
	vR[i] = 2908;
}

}

}
else
{
	if(input[i]>(2905+offset0))
{
	if(input[i]==(2907+offset0))
{
	vR[i] = 2907;
}
else
{
	vR[i] = 2906;
}

}
else
{
	if(input[i]==(2905+offset0))
{
	vR[i] = 2905;
}
else
{
	vR[i] = 2904;
}

}

}

}
else
{
	if(input[i]>(2899+offset0))
{
	if(input[i]>(2901+offset0))
{
	if(input[i]==(2903+offset0))
{
	vR[i] = 2903;
}
else
{
	vR[i] = 2902;
}

}
else
{
	if(input[i]==(2901+offset0))
{
	vR[i] = 2901;
}
else
{
	vR[i] = 2900;
}

}

}
else
{
	if(input[i]>(2897+offset0))
{
	if(input[i]==(2899+offset0))
{
	vR[i] = 2899;
}
else
{
	vR[i] = 2898;
}

}
else
{
	if(input[i]==(2897+offset0))
{
	vR[i] = 2897;
}
else
{
	vR[i] = 2896;
}

}

}

}

}
else
{
	if(input[i]>(2887+offset0))
{
	if(input[i]>(2891+offset0))
{
	if(input[i]>(2893+offset0))
{
	if(input[i]==(2895+offset0))
{
	vR[i] = 2895;
}
else
{
	vR[i] = 2894;
}

}
else
{
	if(input[i]==(2893+offset0))
{
	vR[i] = 2893;
}
else
{
	vR[i] = 2892;
}

}

}
else
{
	if(input[i]>(2889+offset0))
{
	if(input[i]==(2891+offset0))
{
	vR[i] = 2891;
}
else
{
	vR[i] = 2890;
}

}
else
{
	if(input[i]==(2889+offset0))
{
	vR[i] = 2889;
}
else
{
	vR[i] = 2888;
}

}

}

}
else
{
	if(input[i]>(2883+offset0))
{
	if(input[i]>(2885+offset0))
{
	if(input[i]==(2887+offset0))
{
	vR[i] = 2887;
}
else
{
	vR[i] = 2886;
}

}
else
{
	if(input[i]==(2885+offset0))
{
	vR[i] = 2885;
}
else
{
	vR[i] = 2884;
}

}

}
else
{
	if(input[i]>(2881+offset0))
{
	if(input[i]==(2883+offset0))
{
	vR[i] = 2883;
}
else
{
	vR[i] = 2882;
}

}
else
{
	if(input[i]==(2881+offset0))
{
	vR[i] = 2881;
}
else
{
	vR[i] = 2880;
}

}

}

}

}

}

}
else
{
	if(input[i]>(2847+offset0))
{
	if(input[i]>(2863+offset0))
{
	if(input[i]>(2871+offset0))
{
	if(input[i]>(2875+offset0))
{
	if(input[i]>(2877+offset0))
{
	if(input[i]==(2879+offset0))
{
	vR[i] = 2879;
}
else
{
	vR[i] = 2878;
}

}
else
{
	if(input[i]==(2877+offset0))
{
	vR[i] = 2877;
}
else
{
	vR[i] = 2876;
}

}

}
else
{
	if(input[i]>(2873+offset0))
{
	if(input[i]==(2875+offset0))
{
	vR[i] = 2875;
}
else
{
	vR[i] = 2874;
}

}
else
{
	if(input[i]==(2873+offset0))
{
	vR[i] = 2873;
}
else
{
	vR[i] = 2872;
}

}

}

}
else
{
	if(input[i]>(2867+offset0))
{
	if(input[i]>(2869+offset0))
{
	if(input[i]==(2871+offset0))
{
	vR[i] = 2871;
}
else
{
	vR[i] = 2870;
}

}
else
{
	if(input[i]==(2869+offset0))
{
	vR[i] = 2869;
}
else
{
	vR[i] = 2868;
}

}

}
else
{
	if(input[i]>(2865+offset0))
{
	if(input[i]==(2867+offset0))
{
	vR[i] = 2867;
}
else
{
	vR[i] = 2866;
}

}
else
{
	if(input[i]==(2865+offset0))
{
	vR[i] = 2865;
}
else
{
	vR[i] = 2864;
}

}

}

}

}
else
{
	if(input[i]>(2855+offset0))
{
	if(input[i]>(2859+offset0))
{
	if(input[i]>(2861+offset0))
{
	if(input[i]==(2863+offset0))
{
	vR[i] = 2863;
}
else
{
	vR[i] = 2862;
}

}
else
{
	if(input[i]==(2861+offset0))
{
	vR[i] = 2861;
}
else
{
	vR[i] = 2860;
}

}

}
else
{
	if(input[i]>(2857+offset0))
{
	if(input[i]==(2859+offset0))
{
	vR[i] = 2859;
}
else
{
	vR[i] = 2858;
}

}
else
{
	if(input[i]==(2857+offset0))
{
	vR[i] = 2857;
}
else
{
	vR[i] = 2856;
}

}

}

}
else
{
	if(input[i]>(2851+offset0))
{
	if(input[i]>(2853+offset0))
{
	if(input[i]==(2855+offset0))
{
	vR[i] = 2855;
}
else
{
	vR[i] = 2854;
}

}
else
{
	if(input[i]==(2853+offset0))
{
	vR[i] = 2853;
}
else
{
	vR[i] = 2852;
}

}

}
else
{
	if(input[i]>(2849+offset0))
{
	if(input[i]==(2851+offset0))
{
	vR[i] = 2851;
}
else
{
	vR[i] = 2850;
}

}
else
{
	if(input[i]==(2849+offset0))
{
	vR[i] = 2849;
}
else
{
	vR[i] = 2848;
}

}

}

}

}

}
else
{
	if(input[i]>(2831+offset0))
{
	if(input[i]>(2839+offset0))
{
	if(input[i]>(2843+offset0))
{
	if(input[i]>(2845+offset0))
{
	if(input[i]==(2847+offset0))
{
	vR[i] = 2847;
}
else
{
	vR[i] = 2846;
}

}
else
{
	if(input[i]==(2845+offset0))
{
	vR[i] = 2845;
}
else
{
	vR[i] = 2844;
}

}

}
else
{
	if(input[i]>(2841+offset0))
{
	if(input[i]==(2843+offset0))
{
	vR[i] = 2843;
}
else
{
	vR[i] = 2842;
}

}
else
{
	if(input[i]==(2841+offset0))
{
	vR[i] = 2841;
}
else
{
	vR[i] = 2840;
}

}

}

}
else
{
	if(input[i]>(2835+offset0))
{
	if(input[i]>(2837+offset0))
{
	if(input[i]==(2839+offset0))
{
	vR[i] = 2839;
}
else
{
	vR[i] = 2838;
}

}
else
{
	if(input[i]==(2837+offset0))
{
	vR[i] = 2837;
}
else
{
	vR[i] = 2836;
}

}

}
else
{
	if(input[i]>(2833+offset0))
{
	if(input[i]==(2835+offset0))
{
	vR[i] = 2835;
}
else
{
	vR[i] = 2834;
}

}
else
{
	if(input[i]==(2833+offset0))
{
	vR[i] = 2833;
}
else
{
	vR[i] = 2832;
}

}

}

}

}
else
{
	if(input[i]>(2823+offset0))
{
	if(input[i]>(2827+offset0))
{
	if(input[i]>(2829+offset0))
{
	if(input[i]==(2831+offset0))
{
	vR[i] = 2831;
}
else
{
	vR[i] = 2830;
}

}
else
{
	if(input[i]==(2829+offset0))
{
	vR[i] = 2829;
}
else
{
	vR[i] = 2828;
}

}

}
else
{
	if(input[i]>(2825+offset0))
{
	if(input[i]==(2827+offset0))
{
	vR[i] = 2827;
}
else
{
	vR[i] = 2826;
}

}
else
{
	if(input[i]==(2825+offset0))
{
	vR[i] = 2825;
}
else
{
	vR[i] = 2824;
}

}

}

}
else
{
	if(input[i]>(2819+offset0))
{
	if(input[i]>(2821+offset0))
{
	if(input[i]==(2823+offset0))
{
	vR[i] = 2823;
}
else
{
	vR[i] = 2822;
}

}
else
{
	if(input[i]==(2821+offset0))
{
	vR[i] = 2821;
}
else
{
	vR[i] = 2820;
}

}

}
else
{
	if(input[i]>(2817+offset0))
{
	if(input[i]==(2819+offset0))
{
	vR[i] = 2819;
}
else
{
	vR[i] = 2818;
}

}
else
{
	if(input[i]==(2817+offset0))
{
	vR[i] = 2817;
}
else
{
	vR[i] = 2816;
}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(2687+offset0))
{
	if(input[i]>(2751+offset0))
{
	if(input[i]>(2783+offset0))
{
	if(input[i]>(2799+offset0))
{
	if(input[i]>(2807+offset0))
{
	if(input[i]>(2811+offset0))
{
	if(input[i]>(2813+offset0))
{
	if(input[i]==(2815+offset0))
{
	vR[i] = 2815;
}
else
{
	vR[i] = 2814;
}

}
else
{
	if(input[i]==(2813+offset0))
{
	vR[i] = 2813;
}
else
{
	vR[i] = 2812;
}

}

}
else
{
	if(input[i]>(2809+offset0))
{
	if(input[i]==(2811+offset0))
{
	vR[i] = 2811;
}
else
{
	vR[i] = 2810;
}

}
else
{
	if(input[i]==(2809+offset0))
{
	vR[i] = 2809;
}
else
{
	vR[i] = 2808;
}

}

}

}
else
{
	if(input[i]>(2803+offset0))
{
	if(input[i]>(2805+offset0))
{
	if(input[i]==(2807+offset0))
{
	vR[i] = 2807;
}
else
{
	vR[i] = 2806;
}

}
else
{
	if(input[i]==(2805+offset0))
{
	vR[i] = 2805;
}
else
{
	vR[i] = 2804;
}

}

}
else
{
	if(input[i]>(2801+offset0))
{
	if(input[i]==(2803+offset0))
{
	vR[i] = 2803;
}
else
{
	vR[i] = 2802;
}

}
else
{
	if(input[i]==(2801+offset0))
{
	vR[i] = 2801;
}
else
{
	vR[i] = 2800;
}

}

}

}

}
else
{
	if(input[i]>(2791+offset0))
{
	if(input[i]>(2795+offset0))
{
	if(input[i]>(2797+offset0))
{
	if(input[i]==(2799+offset0))
{
	vR[i] = 2799;
}
else
{
	vR[i] = 2798;
}

}
else
{
	if(input[i]==(2797+offset0))
{
	vR[i] = 2797;
}
else
{
	vR[i] = 2796;
}

}

}
else
{
	if(input[i]>(2793+offset0))
{
	if(input[i]==(2795+offset0))
{
	vR[i] = 2795;
}
else
{
	vR[i] = 2794;
}

}
else
{
	if(input[i]==(2793+offset0))
{
	vR[i] = 2793;
}
else
{
	vR[i] = 2792;
}

}

}

}
else
{
	if(input[i]>(2787+offset0))
{
	if(input[i]>(2789+offset0))
{
	if(input[i]==(2791+offset0))
{
	vR[i] = 2791;
}
else
{
	vR[i] = 2790;
}

}
else
{
	if(input[i]==(2789+offset0))
{
	vR[i] = 2789;
}
else
{
	vR[i] = 2788;
}

}

}
else
{
	if(input[i]>(2785+offset0))
{
	if(input[i]==(2787+offset0))
{
	vR[i] = 2787;
}
else
{
	vR[i] = 2786;
}

}
else
{
	if(input[i]==(2785+offset0))
{
	vR[i] = 2785;
}
else
{
	vR[i] = 2784;
}

}

}

}

}

}
else
{
	if(input[i]>(2767+offset0))
{
	if(input[i]>(2775+offset0))
{
	if(input[i]>(2779+offset0))
{
	if(input[i]>(2781+offset0))
{
	if(input[i]==(2783+offset0))
{
	vR[i] = 2783;
}
else
{
	vR[i] = 2782;
}

}
else
{
	if(input[i]==(2781+offset0))
{
	vR[i] = 2781;
}
else
{
	vR[i] = 2780;
}

}

}
else
{
	if(input[i]>(2777+offset0))
{
	if(input[i]==(2779+offset0))
{
	vR[i] = 2779;
}
else
{
	vR[i] = 2778;
}

}
else
{
	if(input[i]==(2777+offset0))
{
	vR[i] = 2777;
}
else
{
	vR[i] = 2776;
}

}

}

}
else
{
	if(input[i]>(2771+offset0))
{
	if(input[i]>(2773+offset0))
{
	if(input[i]==(2775+offset0))
{
	vR[i] = 2775;
}
else
{
	vR[i] = 2774;
}

}
else
{
	if(input[i]==(2773+offset0))
{
	vR[i] = 2773;
}
else
{
	vR[i] = 2772;
}

}

}
else
{
	if(input[i]>(2769+offset0))
{
	if(input[i]==(2771+offset0))
{
	vR[i] = 2771;
}
else
{
	vR[i] = 2770;
}

}
else
{
	if(input[i]==(2769+offset0))
{
	vR[i] = 2769;
}
else
{
	vR[i] = 2768;
}

}

}

}

}
else
{
	if(input[i]>(2759+offset0))
{
	if(input[i]>(2763+offset0))
{
	if(input[i]>(2765+offset0))
{
	if(input[i]==(2767+offset0))
{
	vR[i] = 2767;
}
else
{
	vR[i] = 2766;
}

}
else
{
	if(input[i]==(2765+offset0))
{
	vR[i] = 2765;
}
else
{
	vR[i] = 2764;
}

}

}
else
{
	if(input[i]>(2761+offset0))
{
	if(input[i]==(2763+offset0))
{
	vR[i] = 2763;
}
else
{
	vR[i] = 2762;
}

}
else
{
	if(input[i]==(2761+offset0))
{
	vR[i] = 2761;
}
else
{
	vR[i] = 2760;
}

}

}

}
else
{
	if(input[i]>(2755+offset0))
{
	if(input[i]>(2757+offset0))
{
	if(input[i]==(2759+offset0))
{
	vR[i] = 2759;
}
else
{
	vR[i] = 2758;
}

}
else
{
	if(input[i]==(2757+offset0))
{
	vR[i] = 2757;
}
else
{
	vR[i] = 2756;
}

}

}
else
{
	if(input[i]>(2753+offset0))
{
	if(input[i]==(2755+offset0))
{
	vR[i] = 2755;
}
else
{
	vR[i] = 2754;
}

}
else
{
	if(input[i]==(2753+offset0))
{
	vR[i] = 2753;
}
else
{
	vR[i] = 2752;
}

}

}

}

}

}

}
else
{
	if(input[i]>(2719+offset0))
{
	if(input[i]>(2735+offset0))
{
	if(input[i]>(2743+offset0))
{
	if(input[i]>(2747+offset0))
{
	if(input[i]>(2749+offset0))
{
	if(input[i]==(2751+offset0))
{
	vR[i] = 2751;
}
else
{
	vR[i] = 2750;
}

}
else
{
	if(input[i]==(2749+offset0))
{
	vR[i] = 2749;
}
else
{
	vR[i] = 2748;
}

}

}
else
{
	if(input[i]>(2745+offset0))
{
	if(input[i]==(2747+offset0))
{
	vR[i] = 2747;
}
else
{
	vR[i] = 2746;
}

}
else
{
	if(input[i]==(2745+offset0))
{
	vR[i] = 2745;
}
else
{
	vR[i] = 2744;
}

}

}

}
else
{
	if(input[i]>(2739+offset0))
{
	if(input[i]>(2741+offset0))
{
	if(input[i]==(2743+offset0))
{
	vR[i] = 2743;
}
else
{
	vR[i] = 2742;
}

}
else
{
	if(input[i]==(2741+offset0))
{
	vR[i] = 2741;
}
else
{
	vR[i] = 2740;
}

}

}
else
{
	if(input[i]>(2737+offset0))
{
	if(input[i]==(2739+offset0))
{
	vR[i] = 2739;
}
else
{
	vR[i] = 2738;
}

}
else
{
	if(input[i]==(2737+offset0))
{
	vR[i] = 2737;
}
else
{
	vR[i] = 2736;
}

}

}

}

}
else
{
	if(input[i]>(2727+offset0))
{
	if(input[i]>(2731+offset0))
{
	if(input[i]>(2733+offset0))
{
	if(input[i]==(2735+offset0))
{
	vR[i] = 2735;
}
else
{
	vR[i] = 2734;
}

}
else
{
	if(input[i]==(2733+offset0))
{
	vR[i] = 2733;
}
else
{
	vR[i] = 2732;
}

}

}
else
{
	if(input[i]>(2729+offset0))
{
	if(input[i]==(2731+offset0))
{
	vR[i] = 2731;
}
else
{
	vR[i] = 2730;
}

}
else
{
	if(input[i]==(2729+offset0))
{
	vR[i] = 2729;
}
else
{
	vR[i] = 2728;
}

}

}

}
else
{
	if(input[i]>(2723+offset0))
{
	if(input[i]>(2725+offset0))
{
	if(input[i]==(2727+offset0))
{
	vR[i] = 2727;
}
else
{
	vR[i] = 2726;
}

}
else
{
	if(input[i]==(2725+offset0))
{
	vR[i] = 2725;
}
else
{
	vR[i] = 2724;
}

}

}
else
{
	if(input[i]>(2721+offset0))
{
	if(input[i]==(2723+offset0))
{
	vR[i] = 2723;
}
else
{
	vR[i] = 2722;
}

}
else
{
	if(input[i]==(2721+offset0))
{
	vR[i] = 2721;
}
else
{
	vR[i] = 2720;
}

}

}

}

}

}
else
{
	if(input[i]>(2703+offset0))
{
	if(input[i]>(2711+offset0))
{
	if(input[i]>(2715+offset0))
{
	if(input[i]>(2717+offset0))
{
	if(input[i]==(2719+offset0))
{
	vR[i] = 2719;
}
else
{
	vR[i] = 2718;
}

}
else
{
	if(input[i]==(2717+offset0))
{
	vR[i] = 2717;
}
else
{
	vR[i] = 2716;
}

}

}
else
{
	if(input[i]>(2713+offset0))
{
	if(input[i]==(2715+offset0))
{
	vR[i] = 2715;
}
else
{
	vR[i] = 2714;
}

}
else
{
	if(input[i]==(2713+offset0))
{
	vR[i] = 2713;
}
else
{
	vR[i] = 2712;
}

}

}

}
else
{
	if(input[i]>(2707+offset0))
{
	if(input[i]>(2709+offset0))
{
	if(input[i]==(2711+offset0))
{
	vR[i] = 2711;
}
else
{
	vR[i] = 2710;
}

}
else
{
	if(input[i]==(2709+offset0))
{
	vR[i] = 2709;
}
else
{
	vR[i] = 2708;
}

}

}
else
{
	if(input[i]>(2705+offset0))
{
	if(input[i]==(2707+offset0))
{
	vR[i] = 2707;
}
else
{
	vR[i] = 2706;
}

}
else
{
	if(input[i]==(2705+offset0))
{
	vR[i] = 2705;
}
else
{
	vR[i] = 2704;
}

}

}

}

}
else
{
	if(input[i]>(2695+offset0))
{
	if(input[i]>(2699+offset0))
{
	if(input[i]>(2701+offset0))
{
	if(input[i]==(2703+offset0))
{
	vR[i] = 2703;
}
else
{
	vR[i] = 2702;
}

}
else
{
	if(input[i]==(2701+offset0))
{
	vR[i] = 2701;
}
else
{
	vR[i] = 2700;
}

}

}
else
{
	if(input[i]>(2697+offset0))
{
	if(input[i]==(2699+offset0))
{
	vR[i] = 2699;
}
else
{
	vR[i] = 2698;
}

}
else
{
	if(input[i]==(2697+offset0))
{
	vR[i] = 2697;
}
else
{
	vR[i] = 2696;
}

}

}

}
else
{
	if(input[i]>(2691+offset0))
{
	if(input[i]>(2693+offset0))
{
	if(input[i]==(2695+offset0))
{
	vR[i] = 2695;
}
else
{
	vR[i] = 2694;
}

}
else
{
	if(input[i]==(2693+offset0))
{
	vR[i] = 2693;
}
else
{
	vR[i] = 2692;
}

}

}
else
{
	if(input[i]>(2689+offset0))
{
	if(input[i]==(2691+offset0))
{
	vR[i] = 2691;
}
else
{
	vR[i] = 2690;
}

}
else
{
	if(input[i]==(2689+offset0))
{
	vR[i] = 2689;
}
else
{
	vR[i] = 2688;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(2623+offset0))
{
	if(input[i]>(2655+offset0))
{
	if(input[i]>(2671+offset0))
{
	if(input[i]>(2679+offset0))
{
	if(input[i]>(2683+offset0))
{
	if(input[i]>(2685+offset0))
{
	if(input[i]==(2687+offset0))
{
	vR[i] = 2687;
}
else
{
	vR[i] = 2686;
}

}
else
{
	if(input[i]==(2685+offset0))
{
	vR[i] = 2685;
}
else
{
	vR[i] = 2684;
}

}

}
else
{
	if(input[i]>(2681+offset0))
{
	if(input[i]==(2683+offset0))
{
	vR[i] = 2683;
}
else
{
	vR[i] = 2682;
}

}
else
{
	if(input[i]==(2681+offset0))
{
	vR[i] = 2681;
}
else
{
	vR[i] = 2680;
}

}

}

}
else
{
	if(input[i]>(2675+offset0))
{
	if(input[i]>(2677+offset0))
{
	if(input[i]==(2679+offset0))
{
	vR[i] = 2679;
}
else
{
	vR[i] = 2678;
}

}
else
{
	if(input[i]==(2677+offset0))
{
	vR[i] = 2677;
}
else
{
	vR[i] = 2676;
}

}

}
else
{
	if(input[i]>(2673+offset0))
{
	if(input[i]==(2675+offset0))
{
	vR[i] = 2675;
}
else
{
	vR[i] = 2674;
}

}
else
{
	if(input[i]==(2673+offset0))
{
	vR[i] = 2673;
}
else
{
	vR[i] = 2672;
}

}

}

}

}
else
{
	if(input[i]>(2663+offset0))
{
	if(input[i]>(2667+offset0))
{
	if(input[i]>(2669+offset0))
{
	if(input[i]==(2671+offset0))
{
	vR[i] = 2671;
}
else
{
	vR[i] = 2670;
}

}
else
{
	if(input[i]==(2669+offset0))
{
	vR[i] = 2669;
}
else
{
	vR[i] = 2668;
}

}

}
else
{
	if(input[i]>(2665+offset0))
{
	if(input[i]==(2667+offset0))
{
	vR[i] = 2667;
}
else
{
	vR[i] = 2666;
}

}
else
{
	if(input[i]==(2665+offset0))
{
	vR[i] = 2665;
}
else
{
	vR[i] = 2664;
}

}

}

}
else
{
	if(input[i]>(2659+offset0))
{
	if(input[i]>(2661+offset0))
{
	if(input[i]==(2663+offset0))
{
	vR[i] = 2663;
}
else
{
	vR[i] = 2662;
}

}
else
{
	if(input[i]==(2661+offset0))
{
	vR[i] = 2661;
}
else
{
	vR[i] = 2660;
}

}

}
else
{
	if(input[i]>(2657+offset0))
{
	if(input[i]==(2659+offset0))
{
	vR[i] = 2659;
}
else
{
	vR[i] = 2658;
}

}
else
{
	if(input[i]==(2657+offset0))
{
	vR[i] = 2657;
}
else
{
	vR[i] = 2656;
}

}

}

}

}

}
else
{
	if(input[i]>(2639+offset0))
{
	if(input[i]>(2647+offset0))
{
	if(input[i]>(2651+offset0))
{
	if(input[i]>(2653+offset0))
{
	if(input[i]==(2655+offset0))
{
	vR[i] = 2655;
}
else
{
	vR[i] = 2654;
}

}
else
{
	if(input[i]==(2653+offset0))
{
	vR[i] = 2653;
}
else
{
	vR[i] = 2652;
}

}

}
else
{
	if(input[i]>(2649+offset0))
{
	if(input[i]==(2651+offset0))
{
	vR[i] = 2651;
}
else
{
	vR[i] = 2650;
}

}
else
{
	if(input[i]==(2649+offset0))
{
	vR[i] = 2649;
}
else
{
	vR[i] = 2648;
}

}

}

}
else
{
	if(input[i]>(2643+offset0))
{
	if(input[i]>(2645+offset0))
{
	if(input[i]==(2647+offset0))
{
	vR[i] = 2647;
}
else
{
	vR[i] = 2646;
}

}
else
{
	if(input[i]==(2645+offset0))
{
	vR[i] = 2645;
}
else
{
	vR[i] = 2644;
}

}

}
else
{
	if(input[i]>(2641+offset0))
{
	if(input[i]==(2643+offset0))
{
	vR[i] = 2643;
}
else
{
	vR[i] = 2642;
}

}
else
{
	if(input[i]==(2641+offset0))
{
	vR[i] = 2641;
}
else
{
	vR[i] = 2640;
}

}

}

}

}
else
{
	if(input[i]>(2631+offset0))
{
	if(input[i]>(2635+offset0))
{
	if(input[i]>(2637+offset0))
{
	if(input[i]==(2639+offset0))
{
	vR[i] = 2639;
}
else
{
	vR[i] = 2638;
}

}
else
{
	if(input[i]==(2637+offset0))
{
	vR[i] = 2637;
}
else
{
	vR[i] = 2636;
}

}

}
else
{
	if(input[i]>(2633+offset0))
{
	if(input[i]==(2635+offset0))
{
	vR[i] = 2635;
}
else
{
	vR[i] = 2634;
}

}
else
{
	if(input[i]==(2633+offset0))
{
	vR[i] = 2633;
}
else
{
	vR[i] = 2632;
}

}

}

}
else
{
	if(input[i]>(2627+offset0))
{
	if(input[i]>(2629+offset0))
{
	if(input[i]==(2631+offset0))
{
	vR[i] = 2631;
}
else
{
	vR[i] = 2630;
}

}
else
{
	if(input[i]==(2629+offset0))
{
	vR[i] = 2629;
}
else
{
	vR[i] = 2628;
}

}

}
else
{
	if(input[i]>(2625+offset0))
{
	if(input[i]==(2627+offset0))
{
	vR[i] = 2627;
}
else
{
	vR[i] = 2626;
}

}
else
{
	if(input[i]==(2625+offset0))
{
	vR[i] = 2625;
}
else
{
	vR[i] = 2624;
}

}

}

}

}

}

}
else
{
	if(input[i]>(2591+offset0))
{
	if(input[i]>(2607+offset0))
{
	if(input[i]>(2615+offset0))
{
	if(input[i]>(2619+offset0))
{
	if(input[i]>(2621+offset0))
{
	if(input[i]==(2623+offset0))
{
	vR[i] = 2623;
}
else
{
	vR[i] = 2622;
}

}
else
{
	if(input[i]==(2621+offset0))
{
	vR[i] = 2621;
}
else
{
	vR[i] = 2620;
}

}

}
else
{
	if(input[i]>(2617+offset0))
{
	if(input[i]==(2619+offset0))
{
	vR[i] = 2619;
}
else
{
	vR[i] = 2618;
}

}
else
{
	if(input[i]==(2617+offset0))
{
	vR[i] = 2617;
}
else
{
	vR[i] = 2616;
}

}

}

}
else
{
	if(input[i]>(2611+offset0))
{
	if(input[i]>(2613+offset0))
{
	if(input[i]==(2615+offset0))
{
	vR[i] = 2615;
}
else
{
	vR[i] = 2614;
}

}
else
{
	if(input[i]==(2613+offset0))
{
	vR[i] = 2613;
}
else
{
	vR[i] = 2612;
}

}

}
else
{
	if(input[i]>(2609+offset0))
{
	if(input[i]==(2611+offset0))
{
	vR[i] = 2611;
}
else
{
	vR[i] = 2610;
}

}
else
{
	if(input[i]==(2609+offset0))
{
	vR[i] = 2609;
}
else
{
	vR[i] = 2608;
}

}

}

}

}
else
{
	if(input[i]>(2599+offset0))
{
	if(input[i]>(2603+offset0))
{
	if(input[i]>(2605+offset0))
{
	if(input[i]==(2607+offset0))
{
	vR[i] = 2607;
}
else
{
	vR[i] = 2606;
}

}
else
{
	if(input[i]==(2605+offset0))
{
	vR[i] = 2605;
}
else
{
	vR[i] = 2604;
}

}

}
else
{
	if(input[i]>(2601+offset0))
{
	if(input[i]==(2603+offset0))
{
	vR[i] = 2603;
}
else
{
	vR[i] = 2602;
}

}
else
{
	if(input[i]==(2601+offset0))
{
	vR[i] = 2601;
}
else
{
	vR[i] = 2600;
}

}

}

}
else
{
	if(input[i]>(2595+offset0))
{
	if(input[i]>(2597+offset0))
{
	if(input[i]==(2599+offset0))
{
	vR[i] = 2599;
}
else
{
	vR[i] = 2598;
}

}
else
{
	if(input[i]==(2597+offset0))
{
	vR[i] = 2597;
}
else
{
	vR[i] = 2596;
}

}

}
else
{
	if(input[i]>(2593+offset0))
{
	if(input[i]==(2595+offset0))
{
	vR[i] = 2595;
}
else
{
	vR[i] = 2594;
}

}
else
{
	if(input[i]==(2593+offset0))
{
	vR[i] = 2593;
}
else
{
	vR[i] = 2592;
}

}

}

}

}

}
else
{
	if(input[i]>(2575+offset0))
{
	if(input[i]>(2583+offset0))
{
	if(input[i]>(2587+offset0))
{
	if(input[i]>(2589+offset0))
{
	if(input[i]==(2591+offset0))
{
	vR[i] = 2591;
}
else
{
	vR[i] = 2590;
}

}
else
{
	if(input[i]==(2589+offset0))
{
	vR[i] = 2589;
}
else
{
	vR[i] = 2588;
}

}

}
else
{
	if(input[i]>(2585+offset0))
{
	if(input[i]==(2587+offset0))
{
	vR[i] = 2587;
}
else
{
	vR[i] = 2586;
}

}
else
{
	if(input[i]==(2585+offset0))
{
	vR[i] = 2585;
}
else
{
	vR[i] = 2584;
}

}

}

}
else
{
	if(input[i]>(2579+offset0))
{
	if(input[i]>(2581+offset0))
{
	if(input[i]==(2583+offset0))
{
	vR[i] = 2583;
}
else
{
	vR[i] = 2582;
}

}
else
{
	if(input[i]==(2581+offset0))
{
	vR[i] = 2581;
}
else
{
	vR[i] = 2580;
}

}

}
else
{
	if(input[i]>(2577+offset0))
{
	if(input[i]==(2579+offset0))
{
	vR[i] = 2579;
}
else
{
	vR[i] = 2578;
}

}
else
{
	if(input[i]==(2577+offset0))
{
	vR[i] = 2577;
}
else
{
	vR[i] = 2576;
}

}

}

}

}
else
{
	if(input[i]>(2567+offset0))
{
	if(input[i]>(2571+offset0))
{
	if(input[i]>(2573+offset0))
{
	if(input[i]==(2575+offset0))
{
	vR[i] = 2575;
}
else
{
	vR[i] = 2574;
}

}
else
{
	if(input[i]==(2573+offset0))
{
	vR[i] = 2573;
}
else
{
	vR[i] = 2572;
}

}

}
else
{
	if(input[i]>(2569+offset0))
{
	if(input[i]==(2571+offset0))
{
	vR[i] = 2571;
}
else
{
	vR[i] = 2570;
}

}
else
{
	if(input[i]==(2569+offset0))
{
	vR[i] = 2569;
}
else
{
	vR[i] = 2568;
}

}

}

}
else
{
	if(input[i]>(2563+offset0))
{
	if(input[i]>(2565+offset0))
{
	if(input[i]==(2567+offset0))
{
	vR[i] = 2567;
}
else
{
	vR[i] = 2566;
}

}
else
{
	if(input[i]==(2565+offset0))
{
	vR[i] = 2565;
}
else
{
	vR[i] = 2564;
}

}

}
else
{
	if(input[i]>(2561+offset0))
{
	if(input[i]==(2563+offset0))
{
	vR[i] = 2563;
}
else
{
	vR[i] = 2562;
}

}
else
{
	if(input[i]==(2561+offset0))
{
	vR[i] = 2561;
}
else
{
	vR[i] = 2560;
}

}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(2303+offset0))
{
	if(input[i]>(2431+offset0))
{
	if(input[i]>(2495+offset0))
{
	if(input[i]>(2527+offset0))
{
	if(input[i]>(2543+offset0))
{
	if(input[i]>(2551+offset0))
{
	if(input[i]>(2555+offset0))
{
	if(input[i]>(2557+offset0))
{
	if(input[i]==(2559+offset0))
{
	vR[i] = 2559;
}
else
{
	vR[i] = 2558;
}

}
else
{
	if(input[i]==(2557+offset0))
{
	vR[i] = 2557;
}
else
{
	vR[i] = 2556;
}

}

}
else
{
	if(input[i]>(2553+offset0))
{
	if(input[i]==(2555+offset0))
{
	vR[i] = 2555;
}
else
{
	vR[i] = 2554;
}

}
else
{
	if(input[i]==(2553+offset0))
{
	vR[i] = 2553;
}
else
{
	vR[i] = 2552;
}

}

}

}
else
{
	if(input[i]>(2547+offset0))
{
	if(input[i]>(2549+offset0))
{
	if(input[i]==(2551+offset0))
{
	vR[i] = 2551;
}
else
{
	vR[i] = 2550;
}

}
else
{
	if(input[i]==(2549+offset0))
{
	vR[i] = 2549;
}
else
{
	vR[i] = 2548;
}

}

}
else
{
	if(input[i]>(2545+offset0))
{
	if(input[i]==(2547+offset0))
{
	vR[i] = 2547;
}
else
{
	vR[i] = 2546;
}

}
else
{
	if(input[i]==(2545+offset0))
{
	vR[i] = 2545;
}
else
{
	vR[i] = 2544;
}

}

}

}

}
else
{
	if(input[i]>(2535+offset0))
{
	if(input[i]>(2539+offset0))
{
	if(input[i]>(2541+offset0))
{
	if(input[i]==(2543+offset0))
{
	vR[i] = 2543;
}
else
{
	vR[i] = 2542;
}

}
else
{
	if(input[i]==(2541+offset0))
{
	vR[i] = 2541;
}
else
{
	vR[i] = 2540;
}

}

}
else
{
	if(input[i]>(2537+offset0))
{
	if(input[i]==(2539+offset0))
{
	vR[i] = 2539;
}
else
{
	vR[i] = 2538;
}

}
else
{
	if(input[i]==(2537+offset0))
{
	vR[i] = 2537;
}
else
{
	vR[i] = 2536;
}

}

}

}
else
{
	if(input[i]>(2531+offset0))
{
	if(input[i]>(2533+offset0))
{
	if(input[i]==(2535+offset0))
{
	vR[i] = 2535;
}
else
{
	vR[i] = 2534;
}

}
else
{
	if(input[i]==(2533+offset0))
{
	vR[i] = 2533;
}
else
{
	vR[i] = 2532;
}

}

}
else
{
	if(input[i]>(2529+offset0))
{
	if(input[i]==(2531+offset0))
{
	vR[i] = 2531;
}
else
{
	vR[i] = 2530;
}

}
else
{
	if(input[i]==(2529+offset0))
{
	vR[i] = 2529;
}
else
{
	vR[i] = 2528;
}

}

}

}

}

}
else
{
	if(input[i]>(2511+offset0))
{
	if(input[i]>(2519+offset0))
{
	if(input[i]>(2523+offset0))
{
	if(input[i]>(2525+offset0))
{
	if(input[i]==(2527+offset0))
{
	vR[i] = 2527;
}
else
{
	vR[i] = 2526;
}

}
else
{
	if(input[i]==(2525+offset0))
{
	vR[i] = 2525;
}
else
{
	vR[i] = 2524;
}

}

}
else
{
	if(input[i]>(2521+offset0))
{
	if(input[i]==(2523+offset0))
{
	vR[i] = 2523;
}
else
{
	vR[i] = 2522;
}

}
else
{
	if(input[i]==(2521+offset0))
{
	vR[i] = 2521;
}
else
{
	vR[i] = 2520;
}

}

}

}
else
{
	if(input[i]>(2515+offset0))
{
	if(input[i]>(2517+offset0))
{
	if(input[i]==(2519+offset0))
{
	vR[i] = 2519;
}
else
{
	vR[i] = 2518;
}

}
else
{
	if(input[i]==(2517+offset0))
{
	vR[i] = 2517;
}
else
{
	vR[i] = 2516;
}

}

}
else
{
	if(input[i]>(2513+offset0))
{
	if(input[i]==(2515+offset0))
{
	vR[i] = 2515;
}
else
{
	vR[i] = 2514;
}

}
else
{
	if(input[i]==(2513+offset0))
{
	vR[i] = 2513;
}
else
{
	vR[i] = 2512;
}

}

}

}

}
else
{
	if(input[i]>(2503+offset0))
{
	if(input[i]>(2507+offset0))
{
	if(input[i]>(2509+offset0))
{
	if(input[i]==(2511+offset0))
{
	vR[i] = 2511;
}
else
{
	vR[i] = 2510;
}

}
else
{
	if(input[i]==(2509+offset0))
{
	vR[i] = 2509;
}
else
{
	vR[i] = 2508;
}

}

}
else
{
	if(input[i]>(2505+offset0))
{
	if(input[i]==(2507+offset0))
{
	vR[i] = 2507;
}
else
{
	vR[i] = 2506;
}

}
else
{
	if(input[i]==(2505+offset0))
{
	vR[i] = 2505;
}
else
{
	vR[i] = 2504;
}

}

}

}
else
{
	if(input[i]>(2499+offset0))
{
	if(input[i]>(2501+offset0))
{
	if(input[i]==(2503+offset0))
{
	vR[i] = 2503;
}
else
{
	vR[i] = 2502;
}

}
else
{
	if(input[i]==(2501+offset0))
{
	vR[i] = 2501;
}
else
{
	vR[i] = 2500;
}

}

}
else
{
	if(input[i]>(2497+offset0))
{
	if(input[i]==(2499+offset0))
{
	vR[i] = 2499;
}
else
{
	vR[i] = 2498;
}

}
else
{
	if(input[i]==(2497+offset0))
{
	vR[i] = 2497;
}
else
{
	vR[i] = 2496;
}

}

}

}

}

}

}
else
{
	if(input[i]>(2463+offset0))
{
	if(input[i]>(2479+offset0))
{
	if(input[i]>(2487+offset0))
{
	if(input[i]>(2491+offset0))
{
	if(input[i]>(2493+offset0))
{
	if(input[i]==(2495+offset0))
{
	vR[i] = 2495;
}
else
{
	vR[i] = 2494;
}

}
else
{
	if(input[i]==(2493+offset0))
{
	vR[i] = 2493;
}
else
{
	vR[i] = 2492;
}

}

}
else
{
	if(input[i]>(2489+offset0))
{
	if(input[i]==(2491+offset0))
{
	vR[i] = 2491;
}
else
{
	vR[i] = 2490;
}

}
else
{
	if(input[i]==(2489+offset0))
{
	vR[i] = 2489;
}
else
{
	vR[i] = 2488;
}

}

}

}
else
{
	if(input[i]>(2483+offset0))
{
	if(input[i]>(2485+offset0))
{
	if(input[i]==(2487+offset0))
{
	vR[i] = 2487;
}
else
{
	vR[i] = 2486;
}

}
else
{
	if(input[i]==(2485+offset0))
{
	vR[i] = 2485;
}
else
{
	vR[i] = 2484;
}

}

}
else
{
	if(input[i]>(2481+offset0))
{
	if(input[i]==(2483+offset0))
{
	vR[i] = 2483;
}
else
{
	vR[i] = 2482;
}

}
else
{
	if(input[i]==(2481+offset0))
{
	vR[i] = 2481;
}
else
{
	vR[i] = 2480;
}

}

}

}

}
else
{
	if(input[i]>(2471+offset0))
{
	if(input[i]>(2475+offset0))
{
	if(input[i]>(2477+offset0))
{
	if(input[i]==(2479+offset0))
{
	vR[i] = 2479;
}
else
{
	vR[i] = 2478;
}

}
else
{
	if(input[i]==(2477+offset0))
{
	vR[i] = 2477;
}
else
{
	vR[i] = 2476;
}

}

}
else
{
	if(input[i]>(2473+offset0))
{
	if(input[i]==(2475+offset0))
{
	vR[i] = 2475;
}
else
{
	vR[i] = 2474;
}

}
else
{
	if(input[i]==(2473+offset0))
{
	vR[i] = 2473;
}
else
{
	vR[i] = 2472;
}

}

}

}
else
{
	if(input[i]>(2467+offset0))
{
	if(input[i]>(2469+offset0))
{
	if(input[i]==(2471+offset0))
{
	vR[i] = 2471;
}
else
{
	vR[i] = 2470;
}

}
else
{
	if(input[i]==(2469+offset0))
{
	vR[i] = 2469;
}
else
{
	vR[i] = 2468;
}

}

}
else
{
	if(input[i]>(2465+offset0))
{
	if(input[i]==(2467+offset0))
{
	vR[i] = 2467;
}
else
{
	vR[i] = 2466;
}

}
else
{
	if(input[i]==(2465+offset0))
{
	vR[i] = 2465;
}
else
{
	vR[i] = 2464;
}

}

}

}

}

}
else
{
	if(input[i]>(2447+offset0))
{
	if(input[i]>(2455+offset0))
{
	if(input[i]>(2459+offset0))
{
	if(input[i]>(2461+offset0))
{
	if(input[i]==(2463+offset0))
{
	vR[i] = 2463;
}
else
{
	vR[i] = 2462;
}

}
else
{
	if(input[i]==(2461+offset0))
{
	vR[i] = 2461;
}
else
{
	vR[i] = 2460;
}

}

}
else
{
	if(input[i]>(2457+offset0))
{
	if(input[i]==(2459+offset0))
{
	vR[i] = 2459;
}
else
{
	vR[i] = 2458;
}

}
else
{
	if(input[i]==(2457+offset0))
{
	vR[i] = 2457;
}
else
{
	vR[i] = 2456;
}

}

}

}
else
{
	if(input[i]>(2451+offset0))
{
	if(input[i]>(2453+offset0))
{
	if(input[i]==(2455+offset0))
{
	vR[i] = 2455;
}
else
{
	vR[i] = 2454;
}

}
else
{
	if(input[i]==(2453+offset0))
{
	vR[i] = 2453;
}
else
{
	vR[i] = 2452;
}

}

}
else
{
	if(input[i]>(2449+offset0))
{
	if(input[i]==(2451+offset0))
{
	vR[i] = 2451;
}
else
{
	vR[i] = 2450;
}

}
else
{
	if(input[i]==(2449+offset0))
{
	vR[i] = 2449;
}
else
{
	vR[i] = 2448;
}

}

}

}

}
else
{
	if(input[i]>(2439+offset0))
{
	if(input[i]>(2443+offset0))
{
	if(input[i]>(2445+offset0))
{
	if(input[i]==(2447+offset0))
{
	vR[i] = 2447;
}
else
{
	vR[i] = 2446;
}

}
else
{
	if(input[i]==(2445+offset0))
{
	vR[i] = 2445;
}
else
{
	vR[i] = 2444;
}

}

}
else
{
	if(input[i]>(2441+offset0))
{
	if(input[i]==(2443+offset0))
{
	vR[i] = 2443;
}
else
{
	vR[i] = 2442;
}

}
else
{
	if(input[i]==(2441+offset0))
{
	vR[i] = 2441;
}
else
{
	vR[i] = 2440;
}

}

}

}
else
{
	if(input[i]>(2435+offset0))
{
	if(input[i]>(2437+offset0))
{
	if(input[i]==(2439+offset0))
{
	vR[i] = 2439;
}
else
{
	vR[i] = 2438;
}

}
else
{
	if(input[i]==(2437+offset0))
{
	vR[i] = 2437;
}
else
{
	vR[i] = 2436;
}

}

}
else
{
	if(input[i]>(2433+offset0))
{
	if(input[i]==(2435+offset0))
{
	vR[i] = 2435;
}
else
{
	vR[i] = 2434;
}

}
else
{
	if(input[i]==(2433+offset0))
{
	vR[i] = 2433;
}
else
{
	vR[i] = 2432;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(2367+offset0))
{
	if(input[i]>(2399+offset0))
{
	if(input[i]>(2415+offset0))
{
	if(input[i]>(2423+offset0))
{
	if(input[i]>(2427+offset0))
{
	if(input[i]>(2429+offset0))
{
	if(input[i]==(2431+offset0))
{
	vR[i] = 2431;
}
else
{
	vR[i] = 2430;
}

}
else
{
	if(input[i]==(2429+offset0))
{
	vR[i] = 2429;
}
else
{
	vR[i] = 2428;
}

}

}
else
{
	if(input[i]>(2425+offset0))
{
	if(input[i]==(2427+offset0))
{
	vR[i] = 2427;
}
else
{
	vR[i] = 2426;
}

}
else
{
	if(input[i]==(2425+offset0))
{
	vR[i] = 2425;
}
else
{
	vR[i] = 2424;
}

}

}

}
else
{
	if(input[i]>(2419+offset0))
{
	if(input[i]>(2421+offset0))
{
	if(input[i]==(2423+offset0))
{
	vR[i] = 2423;
}
else
{
	vR[i] = 2422;
}

}
else
{
	if(input[i]==(2421+offset0))
{
	vR[i] = 2421;
}
else
{
	vR[i] = 2420;
}

}

}
else
{
	if(input[i]>(2417+offset0))
{
	if(input[i]==(2419+offset0))
{
	vR[i] = 2419;
}
else
{
	vR[i] = 2418;
}

}
else
{
	if(input[i]==(2417+offset0))
{
	vR[i] = 2417;
}
else
{
	vR[i] = 2416;
}

}

}

}

}
else
{
	if(input[i]>(2407+offset0))
{
	if(input[i]>(2411+offset0))
{
	if(input[i]>(2413+offset0))
{
	if(input[i]==(2415+offset0))
{
	vR[i] = 2415;
}
else
{
	vR[i] = 2414;
}

}
else
{
	if(input[i]==(2413+offset0))
{
	vR[i] = 2413;
}
else
{
	vR[i] = 2412;
}

}

}
else
{
	if(input[i]>(2409+offset0))
{
	if(input[i]==(2411+offset0))
{
	vR[i] = 2411;
}
else
{
	vR[i] = 2410;
}

}
else
{
	if(input[i]==(2409+offset0))
{
	vR[i] = 2409;
}
else
{
	vR[i] = 2408;
}

}

}

}
else
{
	if(input[i]>(2403+offset0))
{
	if(input[i]>(2405+offset0))
{
	if(input[i]==(2407+offset0))
{
	vR[i] = 2407;
}
else
{
	vR[i] = 2406;
}

}
else
{
	if(input[i]==(2405+offset0))
{
	vR[i] = 2405;
}
else
{
	vR[i] = 2404;
}

}

}
else
{
	if(input[i]>(2401+offset0))
{
	if(input[i]==(2403+offset0))
{
	vR[i] = 2403;
}
else
{
	vR[i] = 2402;
}

}
else
{
	if(input[i]==(2401+offset0))
{
	vR[i] = 2401;
}
else
{
	vR[i] = 2400;
}

}

}

}

}

}
else
{
	if(input[i]>(2383+offset0))
{
	if(input[i]>(2391+offset0))
{
	if(input[i]>(2395+offset0))
{
	if(input[i]>(2397+offset0))
{
	if(input[i]==(2399+offset0))
{
	vR[i] = 2399;
}
else
{
	vR[i] = 2398;
}

}
else
{
	if(input[i]==(2397+offset0))
{
	vR[i] = 2397;
}
else
{
	vR[i] = 2396;
}

}

}
else
{
	if(input[i]>(2393+offset0))
{
	if(input[i]==(2395+offset0))
{
	vR[i] = 2395;
}
else
{
	vR[i] = 2394;
}

}
else
{
	if(input[i]==(2393+offset0))
{
	vR[i] = 2393;
}
else
{
	vR[i] = 2392;
}

}

}

}
else
{
	if(input[i]>(2387+offset0))
{
	if(input[i]>(2389+offset0))
{
	if(input[i]==(2391+offset0))
{
	vR[i] = 2391;
}
else
{
	vR[i] = 2390;
}

}
else
{
	if(input[i]==(2389+offset0))
{
	vR[i] = 2389;
}
else
{
	vR[i] = 2388;
}

}

}
else
{
	if(input[i]>(2385+offset0))
{
	if(input[i]==(2387+offset0))
{
	vR[i] = 2387;
}
else
{
	vR[i] = 2386;
}

}
else
{
	if(input[i]==(2385+offset0))
{
	vR[i] = 2385;
}
else
{
	vR[i] = 2384;
}

}

}

}

}
else
{
	if(input[i]>(2375+offset0))
{
	if(input[i]>(2379+offset0))
{
	if(input[i]>(2381+offset0))
{
	if(input[i]==(2383+offset0))
{
	vR[i] = 2383;
}
else
{
	vR[i] = 2382;
}

}
else
{
	if(input[i]==(2381+offset0))
{
	vR[i] = 2381;
}
else
{
	vR[i] = 2380;
}

}

}
else
{
	if(input[i]>(2377+offset0))
{
	if(input[i]==(2379+offset0))
{
	vR[i] = 2379;
}
else
{
	vR[i] = 2378;
}

}
else
{
	if(input[i]==(2377+offset0))
{
	vR[i] = 2377;
}
else
{
	vR[i] = 2376;
}

}

}

}
else
{
	if(input[i]>(2371+offset0))
{
	if(input[i]>(2373+offset0))
{
	if(input[i]==(2375+offset0))
{
	vR[i] = 2375;
}
else
{
	vR[i] = 2374;
}

}
else
{
	if(input[i]==(2373+offset0))
{
	vR[i] = 2373;
}
else
{
	vR[i] = 2372;
}

}

}
else
{
	if(input[i]>(2369+offset0))
{
	if(input[i]==(2371+offset0))
{
	vR[i] = 2371;
}
else
{
	vR[i] = 2370;
}

}
else
{
	if(input[i]==(2369+offset0))
{
	vR[i] = 2369;
}
else
{
	vR[i] = 2368;
}

}

}

}

}

}

}
else
{
	if(input[i]>(2335+offset0))
{
	if(input[i]>(2351+offset0))
{
	if(input[i]>(2359+offset0))
{
	if(input[i]>(2363+offset0))
{
	if(input[i]>(2365+offset0))
{
	if(input[i]==(2367+offset0))
{
	vR[i] = 2367;
}
else
{
	vR[i] = 2366;
}

}
else
{
	if(input[i]==(2365+offset0))
{
	vR[i] = 2365;
}
else
{
	vR[i] = 2364;
}

}

}
else
{
	if(input[i]>(2361+offset0))
{
	if(input[i]==(2363+offset0))
{
	vR[i] = 2363;
}
else
{
	vR[i] = 2362;
}

}
else
{
	if(input[i]==(2361+offset0))
{
	vR[i] = 2361;
}
else
{
	vR[i] = 2360;
}

}

}

}
else
{
	if(input[i]>(2355+offset0))
{
	if(input[i]>(2357+offset0))
{
	if(input[i]==(2359+offset0))
{
	vR[i] = 2359;
}
else
{
	vR[i] = 2358;
}

}
else
{
	if(input[i]==(2357+offset0))
{
	vR[i] = 2357;
}
else
{
	vR[i] = 2356;
}

}

}
else
{
	if(input[i]>(2353+offset0))
{
	if(input[i]==(2355+offset0))
{
	vR[i] = 2355;
}
else
{
	vR[i] = 2354;
}

}
else
{
	if(input[i]==(2353+offset0))
{
	vR[i] = 2353;
}
else
{
	vR[i] = 2352;
}

}

}

}

}
else
{
	if(input[i]>(2343+offset0))
{
	if(input[i]>(2347+offset0))
{
	if(input[i]>(2349+offset0))
{
	if(input[i]==(2351+offset0))
{
	vR[i] = 2351;
}
else
{
	vR[i] = 2350;
}

}
else
{
	if(input[i]==(2349+offset0))
{
	vR[i] = 2349;
}
else
{
	vR[i] = 2348;
}

}

}
else
{
	if(input[i]>(2345+offset0))
{
	if(input[i]==(2347+offset0))
{
	vR[i] = 2347;
}
else
{
	vR[i] = 2346;
}

}
else
{
	if(input[i]==(2345+offset0))
{
	vR[i] = 2345;
}
else
{
	vR[i] = 2344;
}

}

}

}
else
{
	if(input[i]>(2339+offset0))
{
	if(input[i]>(2341+offset0))
{
	if(input[i]==(2343+offset0))
{
	vR[i] = 2343;
}
else
{
	vR[i] = 2342;
}

}
else
{
	if(input[i]==(2341+offset0))
{
	vR[i] = 2341;
}
else
{
	vR[i] = 2340;
}

}

}
else
{
	if(input[i]>(2337+offset0))
{
	if(input[i]==(2339+offset0))
{
	vR[i] = 2339;
}
else
{
	vR[i] = 2338;
}

}
else
{
	if(input[i]==(2337+offset0))
{
	vR[i] = 2337;
}
else
{
	vR[i] = 2336;
}

}

}

}

}

}
else
{
	if(input[i]>(2319+offset0))
{
	if(input[i]>(2327+offset0))
{
	if(input[i]>(2331+offset0))
{
	if(input[i]>(2333+offset0))
{
	if(input[i]==(2335+offset0))
{
	vR[i] = 2335;
}
else
{
	vR[i] = 2334;
}

}
else
{
	if(input[i]==(2333+offset0))
{
	vR[i] = 2333;
}
else
{
	vR[i] = 2332;
}

}

}
else
{
	if(input[i]>(2329+offset0))
{
	if(input[i]==(2331+offset0))
{
	vR[i] = 2331;
}
else
{
	vR[i] = 2330;
}

}
else
{
	if(input[i]==(2329+offset0))
{
	vR[i] = 2329;
}
else
{
	vR[i] = 2328;
}

}

}

}
else
{
	if(input[i]>(2323+offset0))
{
	if(input[i]>(2325+offset0))
{
	if(input[i]==(2327+offset0))
{
	vR[i] = 2327;
}
else
{
	vR[i] = 2326;
}

}
else
{
	if(input[i]==(2325+offset0))
{
	vR[i] = 2325;
}
else
{
	vR[i] = 2324;
}

}

}
else
{
	if(input[i]>(2321+offset0))
{
	if(input[i]==(2323+offset0))
{
	vR[i] = 2323;
}
else
{
	vR[i] = 2322;
}

}
else
{
	if(input[i]==(2321+offset0))
{
	vR[i] = 2321;
}
else
{
	vR[i] = 2320;
}

}

}

}

}
else
{
	if(input[i]>(2311+offset0))
{
	if(input[i]>(2315+offset0))
{
	if(input[i]>(2317+offset0))
{
	if(input[i]==(2319+offset0))
{
	vR[i] = 2319;
}
else
{
	vR[i] = 2318;
}

}
else
{
	if(input[i]==(2317+offset0))
{
	vR[i] = 2317;
}
else
{
	vR[i] = 2316;
}

}

}
else
{
	if(input[i]>(2313+offset0))
{
	if(input[i]==(2315+offset0))
{
	vR[i] = 2315;
}
else
{
	vR[i] = 2314;
}

}
else
{
	if(input[i]==(2313+offset0))
{
	vR[i] = 2313;
}
else
{
	vR[i] = 2312;
}

}

}

}
else
{
	if(input[i]>(2307+offset0))
{
	if(input[i]>(2309+offset0))
{
	if(input[i]==(2311+offset0))
{
	vR[i] = 2311;
}
else
{
	vR[i] = 2310;
}

}
else
{
	if(input[i]==(2309+offset0))
{
	vR[i] = 2309;
}
else
{
	vR[i] = 2308;
}

}

}
else
{
	if(input[i]>(2305+offset0))
{
	if(input[i]==(2307+offset0))
{
	vR[i] = 2307;
}
else
{
	vR[i] = 2306;
}

}
else
{
	if(input[i]==(2305+offset0))
{
	vR[i] = 2305;
}
else
{
	vR[i] = 2304;
}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(2175+offset0))
{
	if(input[i]>(2239+offset0))
{
	if(input[i]>(2271+offset0))
{
	if(input[i]>(2287+offset0))
{
	if(input[i]>(2295+offset0))
{
	if(input[i]>(2299+offset0))
{
	if(input[i]>(2301+offset0))
{
	if(input[i]==(2303+offset0))
{
	vR[i] = 2303;
}
else
{
	vR[i] = 2302;
}

}
else
{
	if(input[i]==(2301+offset0))
{
	vR[i] = 2301;
}
else
{
	vR[i] = 2300;
}

}

}
else
{
	if(input[i]>(2297+offset0))
{
	if(input[i]==(2299+offset0))
{
	vR[i] = 2299;
}
else
{
	vR[i] = 2298;
}

}
else
{
	if(input[i]==(2297+offset0))
{
	vR[i] = 2297;
}
else
{
	vR[i] = 2296;
}

}

}

}
else
{
	if(input[i]>(2291+offset0))
{
	if(input[i]>(2293+offset0))
{
	if(input[i]==(2295+offset0))
{
	vR[i] = 2295;
}
else
{
	vR[i] = 2294;
}

}
else
{
	if(input[i]==(2293+offset0))
{
	vR[i] = 2293;
}
else
{
	vR[i] = 2292;
}

}

}
else
{
	if(input[i]>(2289+offset0))
{
	if(input[i]==(2291+offset0))
{
	vR[i] = 2291;
}
else
{
	vR[i] = 2290;
}

}
else
{
	if(input[i]==(2289+offset0))
{
	vR[i] = 2289;
}
else
{
	vR[i] = 2288;
}

}

}

}

}
else
{
	if(input[i]>(2279+offset0))
{
	if(input[i]>(2283+offset0))
{
	if(input[i]>(2285+offset0))
{
	if(input[i]==(2287+offset0))
{
	vR[i] = 2287;
}
else
{
	vR[i] = 2286;
}

}
else
{
	if(input[i]==(2285+offset0))
{
	vR[i] = 2285;
}
else
{
	vR[i] = 2284;
}

}

}
else
{
	if(input[i]>(2281+offset0))
{
	if(input[i]==(2283+offset0))
{
	vR[i] = 2283;
}
else
{
	vR[i] = 2282;
}

}
else
{
	if(input[i]==(2281+offset0))
{
	vR[i] = 2281;
}
else
{
	vR[i] = 2280;
}

}

}

}
else
{
	if(input[i]>(2275+offset0))
{
	if(input[i]>(2277+offset0))
{
	if(input[i]==(2279+offset0))
{
	vR[i] = 2279;
}
else
{
	vR[i] = 2278;
}

}
else
{
	if(input[i]==(2277+offset0))
{
	vR[i] = 2277;
}
else
{
	vR[i] = 2276;
}

}

}
else
{
	if(input[i]>(2273+offset0))
{
	if(input[i]==(2275+offset0))
{
	vR[i] = 2275;
}
else
{
	vR[i] = 2274;
}

}
else
{
	if(input[i]==(2273+offset0))
{
	vR[i] = 2273;
}
else
{
	vR[i] = 2272;
}

}

}

}

}

}
else
{
	if(input[i]>(2255+offset0))
{
	if(input[i]>(2263+offset0))
{
	if(input[i]>(2267+offset0))
{
	if(input[i]>(2269+offset0))
{
	if(input[i]==(2271+offset0))
{
	vR[i] = 2271;
}
else
{
	vR[i] = 2270;
}

}
else
{
	if(input[i]==(2269+offset0))
{
	vR[i] = 2269;
}
else
{
	vR[i] = 2268;
}

}

}
else
{
	if(input[i]>(2265+offset0))
{
	if(input[i]==(2267+offset0))
{
	vR[i] = 2267;
}
else
{
	vR[i] = 2266;
}

}
else
{
	if(input[i]==(2265+offset0))
{
	vR[i] = 2265;
}
else
{
	vR[i] = 2264;
}

}

}

}
else
{
	if(input[i]>(2259+offset0))
{
	if(input[i]>(2261+offset0))
{
	if(input[i]==(2263+offset0))
{
	vR[i] = 2263;
}
else
{
	vR[i] = 2262;
}

}
else
{
	if(input[i]==(2261+offset0))
{
	vR[i] = 2261;
}
else
{
	vR[i] = 2260;
}

}

}
else
{
	if(input[i]>(2257+offset0))
{
	if(input[i]==(2259+offset0))
{
	vR[i] = 2259;
}
else
{
	vR[i] = 2258;
}

}
else
{
	if(input[i]==(2257+offset0))
{
	vR[i] = 2257;
}
else
{
	vR[i] = 2256;
}

}

}

}

}
else
{
	if(input[i]>(2247+offset0))
{
	if(input[i]>(2251+offset0))
{
	if(input[i]>(2253+offset0))
{
	if(input[i]==(2255+offset0))
{
	vR[i] = 2255;
}
else
{
	vR[i] = 2254;
}

}
else
{
	if(input[i]==(2253+offset0))
{
	vR[i] = 2253;
}
else
{
	vR[i] = 2252;
}

}

}
else
{
	if(input[i]>(2249+offset0))
{
	if(input[i]==(2251+offset0))
{
	vR[i] = 2251;
}
else
{
	vR[i] = 2250;
}

}
else
{
	if(input[i]==(2249+offset0))
{
	vR[i] = 2249;
}
else
{
	vR[i] = 2248;
}

}

}

}
else
{
	if(input[i]>(2243+offset0))
{
	if(input[i]>(2245+offset0))
{
	if(input[i]==(2247+offset0))
{
	vR[i] = 2247;
}
else
{
	vR[i] = 2246;
}

}
else
{
	if(input[i]==(2245+offset0))
{
	vR[i] = 2245;
}
else
{
	vR[i] = 2244;
}

}

}
else
{
	if(input[i]>(2241+offset0))
{
	if(input[i]==(2243+offset0))
{
	vR[i] = 2243;
}
else
{
	vR[i] = 2242;
}

}
else
{
	if(input[i]==(2241+offset0))
{
	vR[i] = 2241;
}
else
{
	vR[i] = 2240;
}

}

}

}

}

}

}
else
{
	if(input[i]>(2207+offset0))
{
	if(input[i]>(2223+offset0))
{
	if(input[i]>(2231+offset0))
{
	if(input[i]>(2235+offset0))
{
	if(input[i]>(2237+offset0))
{
	if(input[i]==(2239+offset0))
{
	vR[i] = 2239;
}
else
{
	vR[i] = 2238;
}

}
else
{
	if(input[i]==(2237+offset0))
{
	vR[i] = 2237;
}
else
{
	vR[i] = 2236;
}

}

}
else
{
	if(input[i]>(2233+offset0))
{
	if(input[i]==(2235+offset0))
{
	vR[i] = 2235;
}
else
{
	vR[i] = 2234;
}

}
else
{
	if(input[i]==(2233+offset0))
{
	vR[i] = 2233;
}
else
{
	vR[i] = 2232;
}

}

}

}
else
{
	if(input[i]>(2227+offset0))
{
	if(input[i]>(2229+offset0))
{
	if(input[i]==(2231+offset0))
{
	vR[i] = 2231;
}
else
{
	vR[i] = 2230;
}

}
else
{
	if(input[i]==(2229+offset0))
{
	vR[i] = 2229;
}
else
{
	vR[i] = 2228;
}

}

}
else
{
	if(input[i]>(2225+offset0))
{
	if(input[i]==(2227+offset0))
{
	vR[i] = 2227;
}
else
{
	vR[i] = 2226;
}

}
else
{
	if(input[i]==(2225+offset0))
{
	vR[i] = 2225;
}
else
{
	vR[i] = 2224;
}

}

}

}

}
else
{
	if(input[i]>(2215+offset0))
{
	if(input[i]>(2219+offset0))
{
	if(input[i]>(2221+offset0))
{
	if(input[i]==(2223+offset0))
{
	vR[i] = 2223;
}
else
{
	vR[i] = 2222;
}

}
else
{
	if(input[i]==(2221+offset0))
{
	vR[i] = 2221;
}
else
{
	vR[i] = 2220;
}

}

}
else
{
	if(input[i]>(2217+offset0))
{
	if(input[i]==(2219+offset0))
{
	vR[i] = 2219;
}
else
{
	vR[i] = 2218;
}

}
else
{
	if(input[i]==(2217+offset0))
{
	vR[i] = 2217;
}
else
{
	vR[i] = 2216;
}

}

}

}
else
{
	if(input[i]>(2211+offset0))
{
	if(input[i]>(2213+offset0))
{
	if(input[i]==(2215+offset0))
{
	vR[i] = 2215;
}
else
{
	vR[i] = 2214;
}

}
else
{
	if(input[i]==(2213+offset0))
{
	vR[i] = 2213;
}
else
{
	vR[i] = 2212;
}

}

}
else
{
	if(input[i]>(2209+offset0))
{
	if(input[i]==(2211+offset0))
{
	vR[i] = 2211;
}
else
{
	vR[i] = 2210;
}

}
else
{
	if(input[i]==(2209+offset0))
{
	vR[i] = 2209;
}
else
{
	vR[i] = 2208;
}

}

}

}

}

}
else
{
	if(input[i]>(2191+offset0))
{
	if(input[i]>(2199+offset0))
{
	if(input[i]>(2203+offset0))
{
	if(input[i]>(2205+offset0))
{
	if(input[i]==(2207+offset0))
{
	vR[i] = 2207;
}
else
{
	vR[i] = 2206;
}

}
else
{
	if(input[i]==(2205+offset0))
{
	vR[i] = 2205;
}
else
{
	vR[i] = 2204;
}

}

}
else
{
	if(input[i]>(2201+offset0))
{
	if(input[i]==(2203+offset0))
{
	vR[i] = 2203;
}
else
{
	vR[i] = 2202;
}

}
else
{
	if(input[i]==(2201+offset0))
{
	vR[i] = 2201;
}
else
{
	vR[i] = 2200;
}

}

}

}
else
{
	if(input[i]>(2195+offset0))
{
	if(input[i]>(2197+offset0))
{
	if(input[i]==(2199+offset0))
{
	vR[i] = 2199;
}
else
{
	vR[i] = 2198;
}

}
else
{
	if(input[i]==(2197+offset0))
{
	vR[i] = 2197;
}
else
{
	vR[i] = 2196;
}

}

}
else
{
	if(input[i]>(2193+offset0))
{
	if(input[i]==(2195+offset0))
{
	vR[i] = 2195;
}
else
{
	vR[i] = 2194;
}

}
else
{
	if(input[i]==(2193+offset0))
{
	vR[i] = 2193;
}
else
{
	vR[i] = 2192;
}

}

}

}

}
else
{
	if(input[i]>(2183+offset0))
{
	if(input[i]>(2187+offset0))
{
	if(input[i]>(2189+offset0))
{
	if(input[i]==(2191+offset0))
{
	vR[i] = 2191;
}
else
{
	vR[i] = 2190;
}

}
else
{
	if(input[i]==(2189+offset0))
{
	vR[i] = 2189;
}
else
{
	vR[i] = 2188;
}

}

}
else
{
	if(input[i]>(2185+offset0))
{
	if(input[i]==(2187+offset0))
{
	vR[i] = 2187;
}
else
{
	vR[i] = 2186;
}

}
else
{
	if(input[i]==(2185+offset0))
{
	vR[i] = 2185;
}
else
{
	vR[i] = 2184;
}

}

}

}
else
{
	if(input[i]>(2179+offset0))
{
	if(input[i]>(2181+offset0))
{
	if(input[i]==(2183+offset0))
{
	vR[i] = 2183;
}
else
{
	vR[i] = 2182;
}

}
else
{
	if(input[i]==(2181+offset0))
{
	vR[i] = 2181;
}
else
{
	vR[i] = 2180;
}

}

}
else
{
	if(input[i]>(2177+offset0))
{
	if(input[i]==(2179+offset0))
{
	vR[i] = 2179;
}
else
{
	vR[i] = 2178;
}

}
else
{
	if(input[i]==(2177+offset0))
{
	vR[i] = 2177;
}
else
{
	vR[i] = 2176;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(2111+offset0))
{
	if(input[i]>(2143+offset0))
{
	if(input[i]>(2159+offset0))
{
	if(input[i]>(2167+offset0))
{
	if(input[i]>(2171+offset0))
{
	if(input[i]>(2173+offset0))
{
	if(input[i]==(2175+offset0))
{
	vR[i] = 2175;
}
else
{
	vR[i] = 2174;
}

}
else
{
	if(input[i]==(2173+offset0))
{
	vR[i] = 2173;
}
else
{
	vR[i] = 2172;
}

}

}
else
{
	if(input[i]>(2169+offset0))
{
	if(input[i]==(2171+offset0))
{
	vR[i] = 2171;
}
else
{
	vR[i] = 2170;
}

}
else
{
	if(input[i]==(2169+offset0))
{
	vR[i] = 2169;
}
else
{
	vR[i] = 2168;
}

}

}

}
else
{
	if(input[i]>(2163+offset0))
{
	if(input[i]>(2165+offset0))
{
	if(input[i]==(2167+offset0))
{
	vR[i] = 2167;
}
else
{
	vR[i] = 2166;
}

}
else
{
	if(input[i]==(2165+offset0))
{
	vR[i] = 2165;
}
else
{
	vR[i] = 2164;
}

}

}
else
{
	if(input[i]>(2161+offset0))
{
	if(input[i]==(2163+offset0))
{
	vR[i] = 2163;
}
else
{
	vR[i] = 2162;
}

}
else
{
	if(input[i]==(2161+offset0))
{
	vR[i] = 2161;
}
else
{
	vR[i] = 2160;
}

}

}

}

}
else
{
	if(input[i]>(2151+offset0))
{
	if(input[i]>(2155+offset0))
{
	if(input[i]>(2157+offset0))
{
	if(input[i]==(2159+offset0))
{
	vR[i] = 2159;
}
else
{
	vR[i] = 2158;
}

}
else
{
	if(input[i]==(2157+offset0))
{
	vR[i] = 2157;
}
else
{
	vR[i] = 2156;
}

}

}
else
{
	if(input[i]>(2153+offset0))
{
	if(input[i]==(2155+offset0))
{
	vR[i] = 2155;
}
else
{
	vR[i] = 2154;
}

}
else
{
	if(input[i]==(2153+offset0))
{
	vR[i] = 2153;
}
else
{
	vR[i] = 2152;
}

}

}

}
else
{
	if(input[i]>(2147+offset0))
{
	if(input[i]>(2149+offset0))
{
	if(input[i]==(2151+offset0))
{
	vR[i] = 2151;
}
else
{
	vR[i] = 2150;
}

}
else
{
	if(input[i]==(2149+offset0))
{
	vR[i] = 2149;
}
else
{
	vR[i] = 2148;
}

}

}
else
{
	if(input[i]>(2145+offset0))
{
	if(input[i]==(2147+offset0))
{
	vR[i] = 2147;
}
else
{
	vR[i] = 2146;
}

}
else
{
	if(input[i]==(2145+offset0))
{
	vR[i] = 2145;
}
else
{
	vR[i] = 2144;
}

}

}

}

}

}
else
{
	if(input[i]>(2127+offset0))
{
	if(input[i]>(2135+offset0))
{
	if(input[i]>(2139+offset0))
{
	if(input[i]>(2141+offset0))
{
	if(input[i]==(2143+offset0))
{
	vR[i] = 2143;
}
else
{
	vR[i] = 2142;
}

}
else
{
	if(input[i]==(2141+offset0))
{
	vR[i] = 2141;
}
else
{
	vR[i] = 2140;
}

}

}
else
{
	if(input[i]>(2137+offset0))
{
	if(input[i]==(2139+offset0))
{
	vR[i] = 2139;
}
else
{
	vR[i] = 2138;
}

}
else
{
	if(input[i]==(2137+offset0))
{
	vR[i] = 2137;
}
else
{
	vR[i] = 2136;
}

}

}

}
else
{
	if(input[i]>(2131+offset0))
{
	if(input[i]>(2133+offset0))
{
	if(input[i]==(2135+offset0))
{
	vR[i] = 2135;
}
else
{
	vR[i] = 2134;
}

}
else
{
	if(input[i]==(2133+offset0))
{
	vR[i] = 2133;
}
else
{
	vR[i] = 2132;
}

}

}
else
{
	if(input[i]>(2129+offset0))
{
	if(input[i]==(2131+offset0))
{
	vR[i] = 2131;
}
else
{
	vR[i] = 2130;
}

}
else
{
	if(input[i]==(2129+offset0))
{
	vR[i] = 2129;
}
else
{
	vR[i] = 2128;
}

}

}

}

}
else
{
	if(input[i]>(2119+offset0))
{
	if(input[i]>(2123+offset0))
{
	if(input[i]>(2125+offset0))
{
	if(input[i]==(2127+offset0))
{
	vR[i] = 2127;
}
else
{
	vR[i] = 2126;
}

}
else
{
	if(input[i]==(2125+offset0))
{
	vR[i] = 2125;
}
else
{
	vR[i] = 2124;
}

}

}
else
{
	if(input[i]>(2121+offset0))
{
	if(input[i]==(2123+offset0))
{
	vR[i] = 2123;
}
else
{
	vR[i] = 2122;
}

}
else
{
	if(input[i]==(2121+offset0))
{
	vR[i] = 2121;
}
else
{
	vR[i] = 2120;
}

}

}

}
else
{
	if(input[i]>(2115+offset0))
{
	if(input[i]>(2117+offset0))
{
	if(input[i]==(2119+offset0))
{
	vR[i] = 2119;
}
else
{
	vR[i] = 2118;
}

}
else
{
	if(input[i]==(2117+offset0))
{
	vR[i] = 2117;
}
else
{
	vR[i] = 2116;
}

}

}
else
{
	if(input[i]>(2113+offset0))
{
	if(input[i]==(2115+offset0))
{
	vR[i] = 2115;
}
else
{
	vR[i] = 2114;
}

}
else
{
	if(input[i]==(2113+offset0))
{
	vR[i] = 2113;
}
else
{
	vR[i] = 2112;
}

}

}

}

}

}

}
else
{
	if(input[i]>(2079+offset0))
{
	if(input[i]>(2095+offset0))
{
	if(input[i]>(2103+offset0))
{
	if(input[i]>(2107+offset0))
{
	if(input[i]>(2109+offset0))
{
	if(input[i]==(2111+offset0))
{
	vR[i] = 2111;
}
else
{
	vR[i] = 2110;
}

}
else
{
	if(input[i]==(2109+offset0))
{
	vR[i] = 2109;
}
else
{
	vR[i] = 2108;
}

}

}
else
{
	if(input[i]>(2105+offset0))
{
	if(input[i]==(2107+offset0))
{
	vR[i] = 2107;
}
else
{
	vR[i] = 2106;
}

}
else
{
	if(input[i]==(2105+offset0))
{
	vR[i] = 2105;
}
else
{
	vR[i] = 2104;
}

}

}

}
else
{
	if(input[i]>(2099+offset0))
{
	if(input[i]>(2101+offset0))
{
	if(input[i]==(2103+offset0))
{
	vR[i] = 2103;
}
else
{
	vR[i] = 2102;
}

}
else
{
	if(input[i]==(2101+offset0))
{
	vR[i] = 2101;
}
else
{
	vR[i] = 2100;
}

}

}
else
{
	if(input[i]>(2097+offset0))
{
	if(input[i]==(2099+offset0))
{
	vR[i] = 2099;
}
else
{
	vR[i] = 2098;
}

}
else
{
	if(input[i]==(2097+offset0))
{
	vR[i] = 2097;
}
else
{
	vR[i] = 2096;
}

}

}

}

}
else
{
	if(input[i]>(2087+offset0))
{
	if(input[i]>(2091+offset0))
{
	if(input[i]>(2093+offset0))
{
	if(input[i]==(2095+offset0))
{
	vR[i] = 2095;
}
else
{
	vR[i] = 2094;
}

}
else
{
	if(input[i]==(2093+offset0))
{
	vR[i] = 2093;
}
else
{
	vR[i] = 2092;
}

}

}
else
{
	if(input[i]>(2089+offset0))
{
	if(input[i]==(2091+offset0))
{
	vR[i] = 2091;
}
else
{
	vR[i] = 2090;
}

}
else
{
	if(input[i]==(2089+offset0))
{
	vR[i] = 2089;
}
else
{
	vR[i] = 2088;
}

}

}

}
else
{
	if(input[i]>(2083+offset0))
{
	if(input[i]>(2085+offset0))
{
	if(input[i]==(2087+offset0))
{
	vR[i] = 2087;
}
else
{
	vR[i] = 2086;
}

}
else
{
	if(input[i]==(2085+offset0))
{
	vR[i] = 2085;
}
else
{
	vR[i] = 2084;
}

}

}
else
{
	if(input[i]>(2081+offset0))
{
	if(input[i]==(2083+offset0))
{
	vR[i] = 2083;
}
else
{
	vR[i] = 2082;
}

}
else
{
	if(input[i]==(2081+offset0))
{
	vR[i] = 2081;
}
else
{
	vR[i] = 2080;
}

}

}

}

}

}
else
{
	if(input[i]>(2063+offset0))
{
	if(input[i]>(2071+offset0))
{
	if(input[i]>(2075+offset0))
{
	if(input[i]>(2077+offset0))
{
	if(input[i]==(2079+offset0))
{
	vR[i] = 2079;
}
else
{
	vR[i] = 2078;
}

}
else
{
	if(input[i]==(2077+offset0))
{
	vR[i] = 2077;
}
else
{
	vR[i] = 2076;
}

}

}
else
{
	if(input[i]>(2073+offset0))
{
	if(input[i]==(2075+offset0))
{
	vR[i] = 2075;
}
else
{
	vR[i] = 2074;
}

}
else
{
	if(input[i]==(2073+offset0))
{
	vR[i] = 2073;
}
else
{
	vR[i] = 2072;
}

}

}

}
else
{
	if(input[i]>(2067+offset0))
{
	if(input[i]>(2069+offset0))
{
	if(input[i]==(2071+offset0))
{
	vR[i] = 2071;
}
else
{
	vR[i] = 2070;
}

}
else
{
	if(input[i]==(2069+offset0))
{
	vR[i] = 2069;
}
else
{
	vR[i] = 2068;
}

}

}
else
{
	if(input[i]>(2065+offset0))
{
	if(input[i]==(2067+offset0))
{
	vR[i] = 2067;
}
else
{
	vR[i] = 2066;
}

}
else
{
	if(input[i]==(2065+offset0))
{
	vR[i] = 2065;
}
else
{
	vR[i] = 2064;
}

}

}

}

}
else
{
	if(input[i]>(2055+offset0))
{
	if(input[i]>(2059+offset0))
{
	if(input[i]>(2061+offset0))
{
	if(input[i]==(2063+offset0))
{
	vR[i] = 2063;
}
else
{
	vR[i] = 2062;
}

}
else
{
	if(input[i]==(2061+offset0))
{
	vR[i] = 2061;
}
else
{
	vR[i] = 2060;
}

}

}
else
{
	if(input[i]>(2057+offset0))
{
	if(input[i]==(2059+offset0))
{
	vR[i] = 2059;
}
else
{
	vR[i] = 2058;
}

}
else
{
	if(input[i]==(2057+offset0))
{
	vR[i] = 2057;
}
else
{
	vR[i] = 2056;
}

}

}

}
else
{
	if(input[i]>(2051+offset0))
{
	if(input[i]>(2053+offset0))
{
	if(input[i]==(2055+offset0))
{
	vR[i] = 2055;
}
else
{
	vR[i] = 2054;
}

}
else
{
	if(input[i]==(2053+offset0))
{
	vR[i] = 2053;
}
else
{
	vR[i] = 2052;
}

}

}
else
{
	if(input[i]>(2049+offset0))
{
	if(input[i]==(2051+offset0))
{
	vR[i] = 2051;
}
else
{
	vR[i] = 2050;
}

}
else
{
	if(input[i]==(2049+offset0))
{
	vR[i] = 2049;
}
else
{
	vR[i] = 2048;
}

}

}

}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(1023+offset0))
{
	if(input[i]>(1535+offset0))
{
	if(input[i]>(1791+offset0))
{
	if(input[i]>(1919+offset0))
{
	if(input[i]>(1983+offset0))
{
	if(input[i]>(2015+offset0))
{
	if(input[i]>(2031+offset0))
{
	if(input[i]>(2039+offset0))
{
	if(input[i]>(2043+offset0))
{
	if(input[i]>(2045+offset0))
{
	if(input[i]==(2047+offset0))
{
	vR[i] = 2047;
}
else
{
	vR[i] = 2046;
}

}
else
{
	if(input[i]==(2045+offset0))
{
	vR[i] = 2045;
}
else
{
	vR[i] = 2044;
}

}

}
else
{
	if(input[i]>(2041+offset0))
{
	if(input[i]==(2043+offset0))
{
	vR[i] = 2043;
}
else
{
	vR[i] = 2042;
}

}
else
{
	if(input[i]==(2041+offset0))
{
	vR[i] = 2041;
}
else
{
	vR[i] = 2040;
}

}

}

}
else
{
	if(input[i]>(2035+offset0))
{
	if(input[i]>(2037+offset0))
{
	if(input[i]==(2039+offset0))
{
	vR[i] = 2039;
}
else
{
	vR[i] = 2038;
}

}
else
{
	if(input[i]==(2037+offset0))
{
	vR[i] = 2037;
}
else
{
	vR[i] = 2036;
}

}

}
else
{
	if(input[i]>(2033+offset0))
{
	if(input[i]==(2035+offset0))
{
	vR[i] = 2035;
}
else
{
	vR[i] = 2034;
}

}
else
{
	if(input[i]==(2033+offset0))
{
	vR[i] = 2033;
}
else
{
	vR[i] = 2032;
}

}

}

}

}
else
{
	if(input[i]>(2023+offset0))
{
	if(input[i]>(2027+offset0))
{
	if(input[i]>(2029+offset0))
{
	if(input[i]==(2031+offset0))
{
	vR[i] = 2031;
}
else
{
	vR[i] = 2030;
}

}
else
{
	if(input[i]==(2029+offset0))
{
	vR[i] = 2029;
}
else
{
	vR[i] = 2028;
}

}

}
else
{
	if(input[i]>(2025+offset0))
{
	if(input[i]==(2027+offset0))
{
	vR[i] = 2027;
}
else
{
	vR[i] = 2026;
}

}
else
{
	if(input[i]==(2025+offset0))
{
	vR[i] = 2025;
}
else
{
	vR[i] = 2024;
}

}

}

}
else
{
	if(input[i]>(2019+offset0))
{
	if(input[i]>(2021+offset0))
{
	if(input[i]==(2023+offset0))
{
	vR[i] = 2023;
}
else
{
	vR[i] = 2022;
}

}
else
{
	if(input[i]==(2021+offset0))
{
	vR[i] = 2021;
}
else
{
	vR[i] = 2020;
}

}

}
else
{
	if(input[i]>(2017+offset0))
{
	if(input[i]==(2019+offset0))
{
	vR[i] = 2019;
}
else
{
	vR[i] = 2018;
}

}
else
{
	if(input[i]==(2017+offset0))
{
	vR[i] = 2017;
}
else
{
	vR[i] = 2016;
}

}

}

}

}

}
else
{
	if(input[i]>(1999+offset0))
{
	if(input[i]>(2007+offset0))
{
	if(input[i]>(2011+offset0))
{
	if(input[i]>(2013+offset0))
{
	if(input[i]==(2015+offset0))
{
	vR[i] = 2015;
}
else
{
	vR[i] = 2014;
}

}
else
{
	if(input[i]==(2013+offset0))
{
	vR[i] = 2013;
}
else
{
	vR[i] = 2012;
}

}

}
else
{
	if(input[i]>(2009+offset0))
{
	if(input[i]==(2011+offset0))
{
	vR[i] = 2011;
}
else
{
	vR[i] = 2010;
}

}
else
{
	if(input[i]==(2009+offset0))
{
	vR[i] = 2009;
}
else
{
	vR[i] = 2008;
}

}

}

}
else
{
	if(input[i]>(2003+offset0))
{
	if(input[i]>(2005+offset0))
{
	if(input[i]==(2007+offset0))
{
	vR[i] = 2007;
}
else
{
	vR[i] = 2006;
}

}
else
{
	if(input[i]==(2005+offset0))
{
	vR[i] = 2005;
}
else
{
	vR[i] = 2004;
}

}

}
else
{
	if(input[i]>(2001+offset0))
{
	if(input[i]==(2003+offset0))
{
	vR[i] = 2003;
}
else
{
	vR[i] = 2002;
}

}
else
{
	if(input[i]==(2001+offset0))
{
	vR[i] = 2001;
}
else
{
	vR[i] = 2000;
}

}

}

}

}
else
{
	if(input[i]>(1991+offset0))
{
	if(input[i]>(1995+offset0))
{
	if(input[i]>(1997+offset0))
{
	if(input[i]==(1999+offset0))
{
	vR[i] = 1999;
}
else
{
	vR[i] = 1998;
}

}
else
{
	if(input[i]==(1997+offset0))
{
	vR[i] = 1997;
}
else
{
	vR[i] = 1996;
}

}

}
else
{
	if(input[i]>(1993+offset0))
{
	if(input[i]==(1995+offset0))
{
	vR[i] = 1995;
}
else
{
	vR[i] = 1994;
}

}
else
{
	if(input[i]==(1993+offset0))
{
	vR[i] = 1993;
}
else
{
	vR[i] = 1992;
}

}

}

}
else
{
	if(input[i]>(1987+offset0))
{
	if(input[i]>(1989+offset0))
{
	if(input[i]==(1991+offset0))
{
	vR[i] = 1991;
}
else
{
	vR[i] = 1990;
}

}
else
{
	if(input[i]==(1989+offset0))
{
	vR[i] = 1989;
}
else
{
	vR[i] = 1988;
}

}

}
else
{
	if(input[i]>(1985+offset0))
{
	if(input[i]==(1987+offset0))
{
	vR[i] = 1987;
}
else
{
	vR[i] = 1986;
}

}
else
{
	if(input[i]==(1985+offset0))
{
	vR[i] = 1985;
}
else
{
	vR[i] = 1984;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1951+offset0))
{
	if(input[i]>(1967+offset0))
{
	if(input[i]>(1975+offset0))
{
	if(input[i]>(1979+offset0))
{
	if(input[i]>(1981+offset0))
{
	if(input[i]==(1983+offset0))
{
	vR[i] = 1983;
}
else
{
	vR[i] = 1982;
}

}
else
{
	if(input[i]==(1981+offset0))
{
	vR[i] = 1981;
}
else
{
	vR[i] = 1980;
}

}

}
else
{
	if(input[i]>(1977+offset0))
{
	if(input[i]==(1979+offset0))
{
	vR[i] = 1979;
}
else
{
	vR[i] = 1978;
}

}
else
{
	if(input[i]==(1977+offset0))
{
	vR[i] = 1977;
}
else
{
	vR[i] = 1976;
}

}

}

}
else
{
	if(input[i]>(1971+offset0))
{
	if(input[i]>(1973+offset0))
{
	if(input[i]==(1975+offset0))
{
	vR[i] = 1975;
}
else
{
	vR[i] = 1974;
}

}
else
{
	if(input[i]==(1973+offset0))
{
	vR[i] = 1973;
}
else
{
	vR[i] = 1972;
}

}

}
else
{
	if(input[i]>(1969+offset0))
{
	if(input[i]==(1971+offset0))
{
	vR[i] = 1971;
}
else
{
	vR[i] = 1970;
}

}
else
{
	if(input[i]==(1969+offset0))
{
	vR[i] = 1969;
}
else
{
	vR[i] = 1968;
}

}

}

}

}
else
{
	if(input[i]>(1959+offset0))
{
	if(input[i]>(1963+offset0))
{
	if(input[i]>(1965+offset0))
{
	if(input[i]==(1967+offset0))
{
	vR[i] = 1967;
}
else
{
	vR[i] = 1966;
}

}
else
{
	if(input[i]==(1965+offset0))
{
	vR[i] = 1965;
}
else
{
	vR[i] = 1964;
}

}

}
else
{
	if(input[i]>(1961+offset0))
{
	if(input[i]==(1963+offset0))
{
	vR[i] = 1963;
}
else
{
	vR[i] = 1962;
}

}
else
{
	if(input[i]==(1961+offset0))
{
	vR[i] = 1961;
}
else
{
	vR[i] = 1960;
}

}

}

}
else
{
	if(input[i]>(1955+offset0))
{
	if(input[i]>(1957+offset0))
{
	if(input[i]==(1959+offset0))
{
	vR[i] = 1959;
}
else
{
	vR[i] = 1958;
}

}
else
{
	if(input[i]==(1957+offset0))
{
	vR[i] = 1957;
}
else
{
	vR[i] = 1956;
}

}

}
else
{
	if(input[i]>(1953+offset0))
{
	if(input[i]==(1955+offset0))
{
	vR[i] = 1955;
}
else
{
	vR[i] = 1954;
}

}
else
{
	if(input[i]==(1953+offset0))
{
	vR[i] = 1953;
}
else
{
	vR[i] = 1952;
}

}

}

}

}

}
else
{
	if(input[i]>(1935+offset0))
{
	if(input[i]>(1943+offset0))
{
	if(input[i]>(1947+offset0))
{
	if(input[i]>(1949+offset0))
{
	if(input[i]==(1951+offset0))
{
	vR[i] = 1951;
}
else
{
	vR[i] = 1950;
}

}
else
{
	if(input[i]==(1949+offset0))
{
	vR[i] = 1949;
}
else
{
	vR[i] = 1948;
}

}

}
else
{
	if(input[i]>(1945+offset0))
{
	if(input[i]==(1947+offset0))
{
	vR[i] = 1947;
}
else
{
	vR[i] = 1946;
}

}
else
{
	if(input[i]==(1945+offset0))
{
	vR[i] = 1945;
}
else
{
	vR[i] = 1944;
}

}

}

}
else
{
	if(input[i]>(1939+offset0))
{
	if(input[i]>(1941+offset0))
{
	if(input[i]==(1943+offset0))
{
	vR[i] = 1943;
}
else
{
	vR[i] = 1942;
}

}
else
{
	if(input[i]==(1941+offset0))
{
	vR[i] = 1941;
}
else
{
	vR[i] = 1940;
}

}

}
else
{
	if(input[i]>(1937+offset0))
{
	if(input[i]==(1939+offset0))
{
	vR[i] = 1939;
}
else
{
	vR[i] = 1938;
}

}
else
{
	if(input[i]==(1937+offset0))
{
	vR[i] = 1937;
}
else
{
	vR[i] = 1936;
}

}

}

}

}
else
{
	if(input[i]>(1927+offset0))
{
	if(input[i]>(1931+offset0))
{
	if(input[i]>(1933+offset0))
{
	if(input[i]==(1935+offset0))
{
	vR[i] = 1935;
}
else
{
	vR[i] = 1934;
}

}
else
{
	if(input[i]==(1933+offset0))
{
	vR[i] = 1933;
}
else
{
	vR[i] = 1932;
}

}

}
else
{
	if(input[i]>(1929+offset0))
{
	if(input[i]==(1931+offset0))
{
	vR[i] = 1931;
}
else
{
	vR[i] = 1930;
}

}
else
{
	if(input[i]==(1929+offset0))
{
	vR[i] = 1929;
}
else
{
	vR[i] = 1928;
}

}

}

}
else
{
	if(input[i]>(1923+offset0))
{
	if(input[i]>(1925+offset0))
{
	if(input[i]==(1927+offset0))
{
	vR[i] = 1927;
}
else
{
	vR[i] = 1926;
}

}
else
{
	if(input[i]==(1925+offset0))
{
	vR[i] = 1925;
}
else
{
	vR[i] = 1924;
}

}

}
else
{
	if(input[i]>(1921+offset0))
{
	if(input[i]==(1923+offset0))
{
	vR[i] = 1923;
}
else
{
	vR[i] = 1922;
}

}
else
{
	if(input[i]==(1921+offset0))
{
	vR[i] = 1921;
}
else
{
	vR[i] = 1920;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(1855+offset0))
{
	if(input[i]>(1887+offset0))
{
	if(input[i]>(1903+offset0))
{
	if(input[i]>(1911+offset0))
{
	if(input[i]>(1915+offset0))
{
	if(input[i]>(1917+offset0))
{
	if(input[i]==(1919+offset0))
{
	vR[i] = 1919;
}
else
{
	vR[i] = 1918;
}

}
else
{
	if(input[i]==(1917+offset0))
{
	vR[i] = 1917;
}
else
{
	vR[i] = 1916;
}

}

}
else
{
	if(input[i]>(1913+offset0))
{
	if(input[i]==(1915+offset0))
{
	vR[i] = 1915;
}
else
{
	vR[i] = 1914;
}

}
else
{
	if(input[i]==(1913+offset0))
{
	vR[i] = 1913;
}
else
{
	vR[i] = 1912;
}

}

}

}
else
{
	if(input[i]>(1907+offset0))
{
	if(input[i]>(1909+offset0))
{
	if(input[i]==(1911+offset0))
{
	vR[i] = 1911;
}
else
{
	vR[i] = 1910;
}

}
else
{
	if(input[i]==(1909+offset0))
{
	vR[i] = 1909;
}
else
{
	vR[i] = 1908;
}

}

}
else
{
	if(input[i]>(1905+offset0))
{
	if(input[i]==(1907+offset0))
{
	vR[i] = 1907;
}
else
{
	vR[i] = 1906;
}

}
else
{
	if(input[i]==(1905+offset0))
{
	vR[i] = 1905;
}
else
{
	vR[i] = 1904;
}

}

}

}

}
else
{
	if(input[i]>(1895+offset0))
{
	if(input[i]>(1899+offset0))
{
	if(input[i]>(1901+offset0))
{
	if(input[i]==(1903+offset0))
{
	vR[i] = 1903;
}
else
{
	vR[i] = 1902;
}

}
else
{
	if(input[i]==(1901+offset0))
{
	vR[i] = 1901;
}
else
{
	vR[i] = 1900;
}

}

}
else
{
	if(input[i]>(1897+offset0))
{
	if(input[i]==(1899+offset0))
{
	vR[i] = 1899;
}
else
{
	vR[i] = 1898;
}

}
else
{
	if(input[i]==(1897+offset0))
{
	vR[i] = 1897;
}
else
{
	vR[i] = 1896;
}

}

}

}
else
{
	if(input[i]>(1891+offset0))
{
	if(input[i]>(1893+offset0))
{
	if(input[i]==(1895+offset0))
{
	vR[i] = 1895;
}
else
{
	vR[i] = 1894;
}

}
else
{
	if(input[i]==(1893+offset0))
{
	vR[i] = 1893;
}
else
{
	vR[i] = 1892;
}

}

}
else
{
	if(input[i]>(1889+offset0))
{
	if(input[i]==(1891+offset0))
{
	vR[i] = 1891;
}
else
{
	vR[i] = 1890;
}

}
else
{
	if(input[i]==(1889+offset0))
{
	vR[i] = 1889;
}
else
{
	vR[i] = 1888;
}

}

}

}

}

}
else
{
	if(input[i]>(1871+offset0))
{
	if(input[i]>(1879+offset0))
{
	if(input[i]>(1883+offset0))
{
	if(input[i]>(1885+offset0))
{
	if(input[i]==(1887+offset0))
{
	vR[i] = 1887;
}
else
{
	vR[i] = 1886;
}

}
else
{
	if(input[i]==(1885+offset0))
{
	vR[i] = 1885;
}
else
{
	vR[i] = 1884;
}

}

}
else
{
	if(input[i]>(1881+offset0))
{
	if(input[i]==(1883+offset0))
{
	vR[i] = 1883;
}
else
{
	vR[i] = 1882;
}

}
else
{
	if(input[i]==(1881+offset0))
{
	vR[i] = 1881;
}
else
{
	vR[i] = 1880;
}

}

}

}
else
{
	if(input[i]>(1875+offset0))
{
	if(input[i]>(1877+offset0))
{
	if(input[i]==(1879+offset0))
{
	vR[i] = 1879;
}
else
{
	vR[i] = 1878;
}

}
else
{
	if(input[i]==(1877+offset0))
{
	vR[i] = 1877;
}
else
{
	vR[i] = 1876;
}

}

}
else
{
	if(input[i]>(1873+offset0))
{
	if(input[i]==(1875+offset0))
{
	vR[i] = 1875;
}
else
{
	vR[i] = 1874;
}

}
else
{
	if(input[i]==(1873+offset0))
{
	vR[i] = 1873;
}
else
{
	vR[i] = 1872;
}

}

}

}

}
else
{
	if(input[i]>(1863+offset0))
{
	if(input[i]>(1867+offset0))
{
	if(input[i]>(1869+offset0))
{
	if(input[i]==(1871+offset0))
{
	vR[i] = 1871;
}
else
{
	vR[i] = 1870;
}

}
else
{
	if(input[i]==(1869+offset0))
{
	vR[i] = 1869;
}
else
{
	vR[i] = 1868;
}

}

}
else
{
	if(input[i]>(1865+offset0))
{
	if(input[i]==(1867+offset0))
{
	vR[i] = 1867;
}
else
{
	vR[i] = 1866;
}

}
else
{
	if(input[i]==(1865+offset0))
{
	vR[i] = 1865;
}
else
{
	vR[i] = 1864;
}

}

}

}
else
{
	if(input[i]>(1859+offset0))
{
	if(input[i]>(1861+offset0))
{
	if(input[i]==(1863+offset0))
{
	vR[i] = 1863;
}
else
{
	vR[i] = 1862;
}

}
else
{
	if(input[i]==(1861+offset0))
{
	vR[i] = 1861;
}
else
{
	vR[i] = 1860;
}

}

}
else
{
	if(input[i]>(1857+offset0))
{
	if(input[i]==(1859+offset0))
{
	vR[i] = 1859;
}
else
{
	vR[i] = 1858;
}

}
else
{
	if(input[i]==(1857+offset0))
{
	vR[i] = 1857;
}
else
{
	vR[i] = 1856;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1823+offset0))
{
	if(input[i]>(1839+offset0))
{
	if(input[i]>(1847+offset0))
{
	if(input[i]>(1851+offset0))
{
	if(input[i]>(1853+offset0))
{
	if(input[i]==(1855+offset0))
{
	vR[i] = 1855;
}
else
{
	vR[i] = 1854;
}

}
else
{
	if(input[i]==(1853+offset0))
{
	vR[i] = 1853;
}
else
{
	vR[i] = 1852;
}

}

}
else
{
	if(input[i]>(1849+offset0))
{
	if(input[i]==(1851+offset0))
{
	vR[i] = 1851;
}
else
{
	vR[i] = 1850;
}

}
else
{
	if(input[i]==(1849+offset0))
{
	vR[i] = 1849;
}
else
{
	vR[i] = 1848;
}

}

}

}
else
{
	if(input[i]>(1843+offset0))
{
	if(input[i]>(1845+offset0))
{
	if(input[i]==(1847+offset0))
{
	vR[i] = 1847;
}
else
{
	vR[i] = 1846;
}

}
else
{
	if(input[i]==(1845+offset0))
{
	vR[i] = 1845;
}
else
{
	vR[i] = 1844;
}

}

}
else
{
	if(input[i]>(1841+offset0))
{
	if(input[i]==(1843+offset0))
{
	vR[i] = 1843;
}
else
{
	vR[i] = 1842;
}

}
else
{
	if(input[i]==(1841+offset0))
{
	vR[i] = 1841;
}
else
{
	vR[i] = 1840;
}

}

}

}

}
else
{
	if(input[i]>(1831+offset0))
{
	if(input[i]>(1835+offset0))
{
	if(input[i]>(1837+offset0))
{
	if(input[i]==(1839+offset0))
{
	vR[i] = 1839;
}
else
{
	vR[i] = 1838;
}

}
else
{
	if(input[i]==(1837+offset0))
{
	vR[i] = 1837;
}
else
{
	vR[i] = 1836;
}

}

}
else
{
	if(input[i]>(1833+offset0))
{
	if(input[i]==(1835+offset0))
{
	vR[i] = 1835;
}
else
{
	vR[i] = 1834;
}

}
else
{
	if(input[i]==(1833+offset0))
{
	vR[i] = 1833;
}
else
{
	vR[i] = 1832;
}

}

}

}
else
{
	if(input[i]>(1827+offset0))
{
	if(input[i]>(1829+offset0))
{
	if(input[i]==(1831+offset0))
{
	vR[i] = 1831;
}
else
{
	vR[i] = 1830;
}

}
else
{
	if(input[i]==(1829+offset0))
{
	vR[i] = 1829;
}
else
{
	vR[i] = 1828;
}

}

}
else
{
	if(input[i]>(1825+offset0))
{
	if(input[i]==(1827+offset0))
{
	vR[i] = 1827;
}
else
{
	vR[i] = 1826;
}

}
else
{
	if(input[i]==(1825+offset0))
{
	vR[i] = 1825;
}
else
{
	vR[i] = 1824;
}

}

}

}

}

}
else
{
	if(input[i]>(1807+offset0))
{
	if(input[i]>(1815+offset0))
{
	if(input[i]>(1819+offset0))
{
	if(input[i]>(1821+offset0))
{
	if(input[i]==(1823+offset0))
{
	vR[i] = 1823;
}
else
{
	vR[i] = 1822;
}

}
else
{
	if(input[i]==(1821+offset0))
{
	vR[i] = 1821;
}
else
{
	vR[i] = 1820;
}

}

}
else
{
	if(input[i]>(1817+offset0))
{
	if(input[i]==(1819+offset0))
{
	vR[i] = 1819;
}
else
{
	vR[i] = 1818;
}

}
else
{
	if(input[i]==(1817+offset0))
{
	vR[i] = 1817;
}
else
{
	vR[i] = 1816;
}

}

}

}
else
{
	if(input[i]>(1811+offset0))
{
	if(input[i]>(1813+offset0))
{
	if(input[i]==(1815+offset0))
{
	vR[i] = 1815;
}
else
{
	vR[i] = 1814;
}

}
else
{
	if(input[i]==(1813+offset0))
{
	vR[i] = 1813;
}
else
{
	vR[i] = 1812;
}

}

}
else
{
	if(input[i]>(1809+offset0))
{
	if(input[i]==(1811+offset0))
{
	vR[i] = 1811;
}
else
{
	vR[i] = 1810;
}

}
else
{
	if(input[i]==(1809+offset0))
{
	vR[i] = 1809;
}
else
{
	vR[i] = 1808;
}

}

}

}

}
else
{
	if(input[i]>(1799+offset0))
{
	if(input[i]>(1803+offset0))
{
	if(input[i]>(1805+offset0))
{
	if(input[i]==(1807+offset0))
{
	vR[i] = 1807;
}
else
{
	vR[i] = 1806;
}

}
else
{
	if(input[i]==(1805+offset0))
{
	vR[i] = 1805;
}
else
{
	vR[i] = 1804;
}

}

}
else
{
	if(input[i]>(1801+offset0))
{
	if(input[i]==(1803+offset0))
{
	vR[i] = 1803;
}
else
{
	vR[i] = 1802;
}

}
else
{
	if(input[i]==(1801+offset0))
{
	vR[i] = 1801;
}
else
{
	vR[i] = 1800;
}

}

}

}
else
{
	if(input[i]>(1795+offset0))
{
	if(input[i]>(1797+offset0))
{
	if(input[i]==(1799+offset0))
{
	vR[i] = 1799;
}
else
{
	vR[i] = 1798;
}

}
else
{
	if(input[i]==(1797+offset0))
{
	vR[i] = 1797;
}
else
{
	vR[i] = 1796;
}

}

}
else
{
	if(input[i]>(1793+offset0))
{
	if(input[i]==(1795+offset0))
{
	vR[i] = 1795;
}
else
{
	vR[i] = 1794;
}

}
else
{
	if(input[i]==(1793+offset0))
{
	vR[i] = 1793;
}
else
{
	vR[i] = 1792;
}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(1663+offset0))
{
	if(input[i]>(1727+offset0))
{
	if(input[i]>(1759+offset0))
{
	if(input[i]>(1775+offset0))
{
	if(input[i]>(1783+offset0))
{
	if(input[i]>(1787+offset0))
{
	if(input[i]>(1789+offset0))
{
	if(input[i]==(1791+offset0))
{
	vR[i] = 1791;
}
else
{
	vR[i] = 1790;
}

}
else
{
	if(input[i]==(1789+offset0))
{
	vR[i] = 1789;
}
else
{
	vR[i] = 1788;
}

}

}
else
{
	if(input[i]>(1785+offset0))
{
	if(input[i]==(1787+offset0))
{
	vR[i] = 1787;
}
else
{
	vR[i] = 1786;
}

}
else
{
	if(input[i]==(1785+offset0))
{
	vR[i] = 1785;
}
else
{
	vR[i] = 1784;
}

}

}

}
else
{
	if(input[i]>(1779+offset0))
{
	if(input[i]>(1781+offset0))
{
	if(input[i]==(1783+offset0))
{
	vR[i] = 1783;
}
else
{
	vR[i] = 1782;
}

}
else
{
	if(input[i]==(1781+offset0))
{
	vR[i] = 1781;
}
else
{
	vR[i] = 1780;
}

}

}
else
{
	if(input[i]>(1777+offset0))
{
	if(input[i]==(1779+offset0))
{
	vR[i] = 1779;
}
else
{
	vR[i] = 1778;
}

}
else
{
	if(input[i]==(1777+offset0))
{
	vR[i] = 1777;
}
else
{
	vR[i] = 1776;
}

}

}

}

}
else
{
	if(input[i]>(1767+offset0))
{
	if(input[i]>(1771+offset0))
{
	if(input[i]>(1773+offset0))
{
	if(input[i]==(1775+offset0))
{
	vR[i] = 1775;
}
else
{
	vR[i] = 1774;
}

}
else
{
	if(input[i]==(1773+offset0))
{
	vR[i] = 1773;
}
else
{
	vR[i] = 1772;
}

}

}
else
{
	if(input[i]>(1769+offset0))
{
	if(input[i]==(1771+offset0))
{
	vR[i] = 1771;
}
else
{
	vR[i] = 1770;
}

}
else
{
	if(input[i]==(1769+offset0))
{
	vR[i] = 1769;
}
else
{
	vR[i] = 1768;
}

}

}

}
else
{
	if(input[i]>(1763+offset0))
{
	if(input[i]>(1765+offset0))
{
	if(input[i]==(1767+offset0))
{
	vR[i] = 1767;
}
else
{
	vR[i] = 1766;
}

}
else
{
	if(input[i]==(1765+offset0))
{
	vR[i] = 1765;
}
else
{
	vR[i] = 1764;
}

}

}
else
{
	if(input[i]>(1761+offset0))
{
	if(input[i]==(1763+offset0))
{
	vR[i] = 1763;
}
else
{
	vR[i] = 1762;
}

}
else
{
	if(input[i]==(1761+offset0))
{
	vR[i] = 1761;
}
else
{
	vR[i] = 1760;
}

}

}

}

}

}
else
{
	if(input[i]>(1743+offset0))
{
	if(input[i]>(1751+offset0))
{
	if(input[i]>(1755+offset0))
{
	if(input[i]>(1757+offset0))
{
	if(input[i]==(1759+offset0))
{
	vR[i] = 1759;
}
else
{
	vR[i] = 1758;
}

}
else
{
	if(input[i]==(1757+offset0))
{
	vR[i] = 1757;
}
else
{
	vR[i] = 1756;
}

}

}
else
{
	if(input[i]>(1753+offset0))
{
	if(input[i]==(1755+offset0))
{
	vR[i] = 1755;
}
else
{
	vR[i] = 1754;
}

}
else
{
	if(input[i]==(1753+offset0))
{
	vR[i] = 1753;
}
else
{
	vR[i] = 1752;
}

}

}

}
else
{
	if(input[i]>(1747+offset0))
{
	if(input[i]>(1749+offset0))
{
	if(input[i]==(1751+offset0))
{
	vR[i] = 1751;
}
else
{
	vR[i] = 1750;
}

}
else
{
	if(input[i]==(1749+offset0))
{
	vR[i] = 1749;
}
else
{
	vR[i] = 1748;
}

}

}
else
{
	if(input[i]>(1745+offset0))
{
	if(input[i]==(1747+offset0))
{
	vR[i] = 1747;
}
else
{
	vR[i] = 1746;
}

}
else
{
	if(input[i]==(1745+offset0))
{
	vR[i] = 1745;
}
else
{
	vR[i] = 1744;
}

}

}

}

}
else
{
	if(input[i]>(1735+offset0))
{
	if(input[i]>(1739+offset0))
{
	if(input[i]>(1741+offset0))
{
	if(input[i]==(1743+offset0))
{
	vR[i] = 1743;
}
else
{
	vR[i] = 1742;
}

}
else
{
	if(input[i]==(1741+offset0))
{
	vR[i] = 1741;
}
else
{
	vR[i] = 1740;
}

}

}
else
{
	if(input[i]>(1737+offset0))
{
	if(input[i]==(1739+offset0))
{
	vR[i] = 1739;
}
else
{
	vR[i] = 1738;
}

}
else
{
	if(input[i]==(1737+offset0))
{
	vR[i] = 1737;
}
else
{
	vR[i] = 1736;
}

}

}

}
else
{
	if(input[i]>(1731+offset0))
{
	if(input[i]>(1733+offset0))
{
	if(input[i]==(1735+offset0))
{
	vR[i] = 1735;
}
else
{
	vR[i] = 1734;
}

}
else
{
	if(input[i]==(1733+offset0))
{
	vR[i] = 1733;
}
else
{
	vR[i] = 1732;
}

}

}
else
{
	if(input[i]>(1729+offset0))
{
	if(input[i]==(1731+offset0))
{
	vR[i] = 1731;
}
else
{
	vR[i] = 1730;
}

}
else
{
	if(input[i]==(1729+offset0))
{
	vR[i] = 1729;
}
else
{
	vR[i] = 1728;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1695+offset0))
{
	if(input[i]>(1711+offset0))
{
	if(input[i]>(1719+offset0))
{
	if(input[i]>(1723+offset0))
{
	if(input[i]>(1725+offset0))
{
	if(input[i]==(1727+offset0))
{
	vR[i] = 1727;
}
else
{
	vR[i] = 1726;
}

}
else
{
	if(input[i]==(1725+offset0))
{
	vR[i] = 1725;
}
else
{
	vR[i] = 1724;
}

}

}
else
{
	if(input[i]>(1721+offset0))
{
	if(input[i]==(1723+offset0))
{
	vR[i] = 1723;
}
else
{
	vR[i] = 1722;
}

}
else
{
	if(input[i]==(1721+offset0))
{
	vR[i] = 1721;
}
else
{
	vR[i] = 1720;
}

}

}

}
else
{
	if(input[i]>(1715+offset0))
{
	if(input[i]>(1717+offset0))
{
	if(input[i]==(1719+offset0))
{
	vR[i] = 1719;
}
else
{
	vR[i] = 1718;
}

}
else
{
	if(input[i]==(1717+offset0))
{
	vR[i] = 1717;
}
else
{
	vR[i] = 1716;
}

}

}
else
{
	if(input[i]>(1713+offset0))
{
	if(input[i]==(1715+offset0))
{
	vR[i] = 1715;
}
else
{
	vR[i] = 1714;
}

}
else
{
	if(input[i]==(1713+offset0))
{
	vR[i] = 1713;
}
else
{
	vR[i] = 1712;
}

}

}

}

}
else
{
	if(input[i]>(1703+offset0))
{
	if(input[i]>(1707+offset0))
{
	if(input[i]>(1709+offset0))
{
	if(input[i]==(1711+offset0))
{
	vR[i] = 1711;
}
else
{
	vR[i] = 1710;
}

}
else
{
	if(input[i]==(1709+offset0))
{
	vR[i] = 1709;
}
else
{
	vR[i] = 1708;
}

}

}
else
{
	if(input[i]>(1705+offset0))
{
	if(input[i]==(1707+offset0))
{
	vR[i] = 1707;
}
else
{
	vR[i] = 1706;
}

}
else
{
	if(input[i]==(1705+offset0))
{
	vR[i] = 1705;
}
else
{
	vR[i] = 1704;
}

}

}

}
else
{
	if(input[i]>(1699+offset0))
{
	if(input[i]>(1701+offset0))
{
	if(input[i]==(1703+offset0))
{
	vR[i] = 1703;
}
else
{
	vR[i] = 1702;
}

}
else
{
	if(input[i]==(1701+offset0))
{
	vR[i] = 1701;
}
else
{
	vR[i] = 1700;
}

}

}
else
{
	if(input[i]>(1697+offset0))
{
	if(input[i]==(1699+offset0))
{
	vR[i] = 1699;
}
else
{
	vR[i] = 1698;
}

}
else
{
	if(input[i]==(1697+offset0))
{
	vR[i] = 1697;
}
else
{
	vR[i] = 1696;
}

}

}

}

}

}
else
{
	if(input[i]>(1679+offset0))
{
	if(input[i]>(1687+offset0))
{
	if(input[i]>(1691+offset0))
{
	if(input[i]>(1693+offset0))
{
	if(input[i]==(1695+offset0))
{
	vR[i] = 1695;
}
else
{
	vR[i] = 1694;
}

}
else
{
	if(input[i]==(1693+offset0))
{
	vR[i] = 1693;
}
else
{
	vR[i] = 1692;
}

}

}
else
{
	if(input[i]>(1689+offset0))
{
	if(input[i]==(1691+offset0))
{
	vR[i] = 1691;
}
else
{
	vR[i] = 1690;
}

}
else
{
	if(input[i]==(1689+offset0))
{
	vR[i] = 1689;
}
else
{
	vR[i] = 1688;
}

}

}

}
else
{
	if(input[i]>(1683+offset0))
{
	if(input[i]>(1685+offset0))
{
	if(input[i]==(1687+offset0))
{
	vR[i] = 1687;
}
else
{
	vR[i] = 1686;
}

}
else
{
	if(input[i]==(1685+offset0))
{
	vR[i] = 1685;
}
else
{
	vR[i] = 1684;
}

}

}
else
{
	if(input[i]>(1681+offset0))
{
	if(input[i]==(1683+offset0))
{
	vR[i] = 1683;
}
else
{
	vR[i] = 1682;
}

}
else
{
	if(input[i]==(1681+offset0))
{
	vR[i] = 1681;
}
else
{
	vR[i] = 1680;
}

}

}

}

}
else
{
	if(input[i]>(1671+offset0))
{
	if(input[i]>(1675+offset0))
{
	if(input[i]>(1677+offset0))
{
	if(input[i]==(1679+offset0))
{
	vR[i] = 1679;
}
else
{
	vR[i] = 1678;
}

}
else
{
	if(input[i]==(1677+offset0))
{
	vR[i] = 1677;
}
else
{
	vR[i] = 1676;
}

}

}
else
{
	if(input[i]>(1673+offset0))
{
	if(input[i]==(1675+offset0))
{
	vR[i] = 1675;
}
else
{
	vR[i] = 1674;
}

}
else
{
	if(input[i]==(1673+offset0))
{
	vR[i] = 1673;
}
else
{
	vR[i] = 1672;
}

}

}

}
else
{
	if(input[i]>(1667+offset0))
{
	if(input[i]>(1669+offset0))
{
	if(input[i]==(1671+offset0))
{
	vR[i] = 1671;
}
else
{
	vR[i] = 1670;
}

}
else
{
	if(input[i]==(1669+offset0))
{
	vR[i] = 1669;
}
else
{
	vR[i] = 1668;
}

}

}
else
{
	if(input[i]>(1665+offset0))
{
	if(input[i]==(1667+offset0))
{
	vR[i] = 1667;
}
else
{
	vR[i] = 1666;
}

}
else
{
	if(input[i]==(1665+offset0))
{
	vR[i] = 1665;
}
else
{
	vR[i] = 1664;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(1599+offset0))
{
	if(input[i]>(1631+offset0))
{
	if(input[i]>(1647+offset0))
{
	if(input[i]>(1655+offset0))
{
	if(input[i]>(1659+offset0))
{
	if(input[i]>(1661+offset0))
{
	if(input[i]==(1663+offset0))
{
	vR[i] = 1663;
}
else
{
	vR[i] = 1662;
}

}
else
{
	if(input[i]==(1661+offset0))
{
	vR[i] = 1661;
}
else
{
	vR[i] = 1660;
}

}

}
else
{
	if(input[i]>(1657+offset0))
{
	if(input[i]==(1659+offset0))
{
	vR[i] = 1659;
}
else
{
	vR[i] = 1658;
}

}
else
{
	if(input[i]==(1657+offset0))
{
	vR[i] = 1657;
}
else
{
	vR[i] = 1656;
}

}

}

}
else
{
	if(input[i]>(1651+offset0))
{
	if(input[i]>(1653+offset0))
{
	if(input[i]==(1655+offset0))
{
	vR[i] = 1655;
}
else
{
	vR[i] = 1654;
}

}
else
{
	if(input[i]==(1653+offset0))
{
	vR[i] = 1653;
}
else
{
	vR[i] = 1652;
}

}

}
else
{
	if(input[i]>(1649+offset0))
{
	if(input[i]==(1651+offset0))
{
	vR[i] = 1651;
}
else
{
	vR[i] = 1650;
}

}
else
{
	if(input[i]==(1649+offset0))
{
	vR[i] = 1649;
}
else
{
	vR[i] = 1648;
}

}

}

}

}
else
{
	if(input[i]>(1639+offset0))
{
	if(input[i]>(1643+offset0))
{
	if(input[i]>(1645+offset0))
{
	if(input[i]==(1647+offset0))
{
	vR[i] = 1647;
}
else
{
	vR[i] = 1646;
}

}
else
{
	if(input[i]==(1645+offset0))
{
	vR[i] = 1645;
}
else
{
	vR[i] = 1644;
}

}

}
else
{
	if(input[i]>(1641+offset0))
{
	if(input[i]==(1643+offset0))
{
	vR[i] = 1643;
}
else
{
	vR[i] = 1642;
}

}
else
{
	if(input[i]==(1641+offset0))
{
	vR[i] = 1641;
}
else
{
	vR[i] = 1640;
}

}

}

}
else
{
	if(input[i]>(1635+offset0))
{
	if(input[i]>(1637+offset0))
{
	if(input[i]==(1639+offset0))
{
	vR[i] = 1639;
}
else
{
	vR[i] = 1638;
}

}
else
{
	if(input[i]==(1637+offset0))
{
	vR[i] = 1637;
}
else
{
	vR[i] = 1636;
}

}

}
else
{
	if(input[i]>(1633+offset0))
{
	if(input[i]==(1635+offset0))
{
	vR[i] = 1635;
}
else
{
	vR[i] = 1634;
}

}
else
{
	if(input[i]==(1633+offset0))
{
	vR[i] = 1633;
}
else
{
	vR[i] = 1632;
}

}

}

}

}

}
else
{
	if(input[i]>(1615+offset0))
{
	if(input[i]>(1623+offset0))
{
	if(input[i]>(1627+offset0))
{
	if(input[i]>(1629+offset0))
{
	if(input[i]==(1631+offset0))
{
	vR[i] = 1631;
}
else
{
	vR[i] = 1630;
}

}
else
{
	if(input[i]==(1629+offset0))
{
	vR[i] = 1629;
}
else
{
	vR[i] = 1628;
}

}

}
else
{
	if(input[i]>(1625+offset0))
{
	if(input[i]==(1627+offset0))
{
	vR[i] = 1627;
}
else
{
	vR[i] = 1626;
}

}
else
{
	if(input[i]==(1625+offset0))
{
	vR[i] = 1625;
}
else
{
	vR[i] = 1624;
}

}

}

}
else
{
	if(input[i]>(1619+offset0))
{
	if(input[i]>(1621+offset0))
{
	if(input[i]==(1623+offset0))
{
	vR[i] = 1623;
}
else
{
	vR[i] = 1622;
}

}
else
{
	if(input[i]==(1621+offset0))
{
	vR[i] = 1621;
}
else
{
	vR[i] = 1620;
}

}

}
else
{
	if(input[i]>(1617+offset0))
{
	if(input[i]==(1619+offset0))
{
	vR[i] = 1619;
}
else
{
	vR[i] = 1618;
}

}
else
{
	if(input[i]==(1617+offset0))
{
	vR[i] = 1617;
}
else
{
	vR[i] = 1616;
}

}

}

}

}
else
{
	if(input[i]>(1607+offset0))
{
	if(input[i]>(1611+offset0))
{
	if(input[i]>(1613+offset0))
{
	if(input[i]==(1615+offset0))
{
	vR[i] = 1615;
}
else
{
	vR[i] = 1614;
}

}
else
{
	if(input[i]==(1613+offset0))
{
	vR[i] = 1613;
}
else
{
	vR[i] = 1612;
}

}

}
else
{
	if(input[i]>(1609+offset0))
{
	if(input[i]==(1611+offset0))
{
	vR[i] = 1611;
}
else
{
	vR[i] = 1610;
}

}
else
{
	if(input[i]==(1609+offset0))
{
	vR[i] = 1609;
}
else
{
	vR[i] = 1608;
}

}

}

}
else
{
	if(input[i]>(1603+offset0))
{
	if(input[i]>(1605+offset0))
{
	if(input[i]==(1607+offset0))
{
	vR[i] = 1607;
}
else
{
	vR[i] = 1606;
}

}
else
{
	if(input[i]==(1605+offset0))
{
	vR[i] = 1605;
}
else
{
	vR[i] = 1604;
}

}

}
else
{
	if(input[i]>(1601+offset0))
{
	if(input[i]==(1603+offset0))
{
	vR[i] = 1603;
}
else
{
	vR[i] = 1602;
}

}
else
{
	if(input[i]==(1601+offset0))
{
	vR[i] = 1601;
}
else
{
	vR[i] = 1600;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1567+offset0))
{
	if(input[i]>(1583+offset0))
{
	if(input[i]>(1591+offset0))
{
	if(input[i]>(1595+offset0))
{
	if(input[i]>(1597+offset0))
{
	if(input[i]==(1599+offset0))
{
	vR[i] = 1599;
}
else
{
	vR[i] = 1598;
}

}
else
{
	if(input[i]==(1597+offset0))
{
	vR[i] = 1597;
}
else
{
	vR[i] = 1596;
}

}

}
else
{
	if(input[i]>(1593+offset0))
{
	if(input[i]==(1595+offset0))
{
	vR[i] = 1595;
}
else
{
	vR[i] = 1594;
}

}
else
{
	if(input[i]==(1593+offset0))
{
	vR[i] = 1593;
}
else
{
	vR[i] = 1592;
}

}

}

}
else
{
	if(input[i]>(1587+offset0))
{
	if(input[i]>(1589+offset0))
{
	if(input[i]==(1591+offset0))
{
	vR[i] = 1591;
}
else
{
	vR[i] = 1590;
}

}
else
{
	if(input[i]==(1589+offset0))
{
	vR[i] = 1589;
}
else
{
	vR[i] = 1588;
}

}

}
else
{
	if(input[i]>(1585+offset0))
{
	if(input[i]==(1587+offset0))
{
	vR[i] = 1587;
}
else
{
	vR[i] = 1586;
}

}
else
{
	if(input[i]==(1585+offset0))
{
	vR[i] = 1585;
}
else
{
	vR[i] = 1584;
}

}

}

}

}
else
{
	if(input[i]>(1575+offset0))
{
	if(input[i]>(1579+offset0))
{
	if(input[i]>(1581+offset0))
{
	if(input[i]==(1583+offset0))
{
	vR[i] = 1583;
}
else
{
	vR[i] = 1582;
}

}
else
{
	if(input[i]==(1581+offset0))
{
	vR[i] = 1581;
}
else
{
	vR[i] = 1580;
}

}

}
else
{
	if(input[i]>(1577+offset0))
{
	if(input[i]==(1579+offset0))
{
	vR[i] = 1579;
}
else
{
	vR[i] = 1578;
}

}
else
{
	if(input[i]==(1577+offset0))
{
	vR[i] = 1577;
}
else
{
	vR[i] = 1576;
}

}

}

}
else
{
	if(input[i]>(1571+offset0))
{
	if(input[i]>(1573+offset0))
{
	if(input[i]==(1575+offset0))
{
	vR[i] = 1575;
}
else
{
	vR[i] = 1574;
}

}
else
{
	if(input[i]==(1573+offset0))
{
	vR[i] = 1573;
}
else
{
	vR[i] = 1572;
}

}

}
else
{
	if(input[i]>(1569+offset0))
{
	if(input[i]==(1571+offset0))
{
	vR[i] = 1571;
}
else
{
	vR[i] = 1570;
}

}
else
{
	if(input[i]==(1569+offset0))
{
	vR[i] = 1569;
}
else
{
	vR[i] = 1568;
}

}

}

}

}

}
else
{
	if(input[i]>(1551+offset0))
{
	if(input[i]>(1559+offset0))
{
	if(input[i]>(1563+offset0))
{
	if(input[i]>(1565+offset0))
{
	if(input[i]==(1567+offset0))
{
	vR[i] = 1567;
}
else
{
	vR[i] = 1566;
}

}
else
{
	if(input[i]==(1565+offset0))
{
	vR[i] = 1565;
}
else
{
	vR[i] = 1564;
}

}

}
else
{
	if(input[i]>(1561+offset0))
{
	if(input[i]==(1563+offset0))
{
	vR[i] = 1563;
}
else
{
	vR[i] = 1562;
}

}
else
{
	if(input[i]==(1561+offset0))
{
	vR[i] = 1561;
}
else
{
	vR[i] = 1560;
}

}

}

}
else
{
	if(input[i]>(1555+offset0))
{
	if(input[i]>(1557+offset0))
{
	if(input[i]==(1559+offset0))
{
	vR[i] = 1559;
}
else
{
	vR[i] = 1558;
}

}
else
{
	if(input[i]==(1557+offset0))
{
	vR[i] = 1557;
}
else
{
	vR[i] = 1556;
}

}

}
else
{
	if(input[i]>(1553+offset0))
{
	if(input[i]==(1555+offset0))
{
	vR[i] = 1555;
}
else
{
	vR[i] = 1554;
}

}
else
{
	if(input[i]==(1553+offset0))
{
	vR[i] = 1553;
}
else
{
	vR[i] = 1552;
}

}

}

}

}
else
{
	if(input[i]>(1543+offset0))
{
	if(input[i]>(1547+offset0))
{
	if(input[i]>(1549+offset0))
{
	if(input[i]==(1551+offset0))
{
	vR[i] = 1551;
}
else
{
	vR[i] = 1550;
}

}
else
{
	if(input[i]==(1549+offset0))
{
	vR[i] = 1549;
}
else
{
	vR[i] = 1548;
}

}

}
else
{
	if(input[i]>(1545+offset0))
{
	if(input[i]==(1547+offset0))
{
	vR[i] = 1547;
}
else
{
	vR[i] = 1546;
}

}
else
{
	if(input[i]==(1545+offset0))
{
	vR[i] = 1545;
}
else
{
	vR[i] = 1544;
}

}

}

}
else
{
	if(input[i]>(1539+offset0))
{
	if(input[i]>(1541+offset0))
{
	if(input[i]==(1543+offset0))
{
	vR[i] = 1543;
}
else
{
	vR[i] = 1542;
}

}
else
{
	if(input[i]==(1541+offset0))
{
	vR[i] = 1541;
}
else
{
	vR[i] = 1540;
}

}

}
else
{
	if(input[i]>(1537+offset0))
{
	if(input[i]==(1539+offset0))
{
	vR[i] = 1539;
}
else
{
	vR[i] = 1538;
}

}
else
{
	if(input[i]==(1537+offset0))
{
	vR[i] = 1537;
}
else
{
	vR[i] = 1536;
}

}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(1279+offset0))
{
	if(input[i]>(1407+offset0))
{
	if(input[i]>(1471+offset0))
{
	if(input[i]>(1503+offset0))
{
	if(input[i]>(1519+offset0))
{
	if(input[i]>(1527+offset0))
{
	if(input[i]>(1531+offset0))
{
	if(input[i]>(1533+offset0))
{
	if(input[i]==(1535+offset0))
{
	vR[i] = 1535;
}
else
{
	vR[i] = 1534;
}

}
else
{
	if(input[i]==(1533+offset0))
{
	vR[i] = 1533;
}
else
{
	vR[i] = 1532;
}

}

}
else
{
	if(input[i]>(1529+offset0))
{
	if(input[i]==(1531+offset0))
{
	vR[i] = 1531;
}
else
{
	vR[i] = 1530;
}

}
else
{
	if(input[i]==(1529+offset0))
{
	vR[i] = 1529;
}
else
{
	vR[i] = 1528;
}

}

}

}
else
{
	if(input[i]>(1523+offset0))
{
	if(input[i]>(1525+offset0))
{
	if(input[i]==(1527+offset0))
{
	vR[i] = 1527;
}
else
{
	vR[i] = 1526;
}

}
else
{
	if(input[i]==(1525+offset0))
{
	vR[i] = 1525;
}
else
{
	vR[i] = 1524;
}

}

}
else
{
	if(input[i]>(1521+offset0))
{
	if(input[i]==(1523+offset0))
{
	vR[i] = 1523;
}
else
{
	vR[i] = 1522;
}

}
else
{
	if(input[i]==(1521+offset0))
{
	vR[i] = 1521;
}
else
{
	vR[i] = 1520;
}

}

}

}

}
else
{
	if(input[i]>(1511+offset0))
{
	if(input[i]>(1515+offset0))
{
	if(input[i]>(1517+offset0))
{
	if(input[i]==(1519+offset0))
{
	vR[i] = 1519;
}
else
{
	vR[i] = 1518;
}

}
else
{
	if(input[i]==(1517+offset0))
{
	vR[i] = 1517;
}
else
{
	vR[i] = 1516;
}

}

}
else
{
	if(input[i]>(1513+offset0))
{
	if(input[i]==(1515+offset0))
{
	vR[i] = 1515;
}
else
{
	vR[i] = 1514;
}

}
else
{
	if(input[i]==(1513+offset0))
{
	vR[i] = 1513;
}
else
{
	vR[i] = 1512;
}

}

}

}
else
{
	if(input[i]>(1507+offset0))
{
	if(input[i]>(1509+offset0))
{
	if(input[i]==(1511+offset0))
{
	vR[i] = 1511;
}
else
{
	vR[i] = 1510;
}

}
else
{
	if(input[i]==(1509+offset0))
{
	vR[i] = 1509;
}
else
{
	vR[i] = 1508;
}

}

}
else
{
	if(input[i]>(1505+offset0))
{
	if(input[i]==(1507+offset0))
{
	vR[i] = 1507;
}
else
{
	vR[i] = 1506;
}

}
else
{
	if(input[i]==(1505+offset0))
{
	vR[i] = 1505;
}
else
{
	vR[i] = 1504;
}

}

}

}

}

}
else
{
	if(input[i]>(1487+offset0))
{
	if(input[i]>(1495+offset0))
{
	if(input[i]>(1499+offset0))
{
	if(input[i]>(1501+offset0))
{
	if(input[i]==(1503+offset0))
{
	vR[i] = 1503;
}
else
{
	vR[i] = 1502;
}

}
else
{
	if(input[i]==(1501+offset0))
{
	vR[i] = 1501;
}
else
{
	vR[i] = 1500;
}

}

}
else
{
	if(input[i]>(1497+offset0))
{
	if(input[i]==(1499+offset0))
{
	vR[i] = 1499;
}
else
{
	vR[i] = 1498;
}

}
else
{
	if(input[i]==(1497+offset0))
{
	vR[i] = 1497;
}
else
{
	vR[i] = 1496;
}

}

}

}
else
{
	if(input[i]>(1491+offset0))
{
	if(input[i]>(1493+offset0))
{
	if(input[i]==(1495+offset0))
{
	vR[i] = 1495;
}
else
{
	vR[i] = 1494;
}

}
else
{
	if(input[i]==(1493+offset0))
{
	vR[i] = 1493;
}
else
{
	vR[i] = 1492;
}

}

}
else
{
	if(input[i]>(1489+offset0))
{
	if(input[i]==(1491+offset0))
{
	vR[i] = 1491;
}
else
{
	vR[i] = 1490;
}

}
else
{
	if(input[i]==(1489+offset0))
{
	vR[i] = 1489;
}
else
{
	vR[i] = 1488;
}

}

}

}

}
else
{
	if(input[i]>(1479+offset0))
{
	if(input[i]>(1483+offset0))
{
	if(input[i]>(1485+offset0))
{
	if(input[i]==(1487+offset0))
{
	vR[i] = 1487;
}
else
{
	vR[i] = 1486;
}

}
else
{
	if(input[i]==(1485+offset0))
{
	vR[i] = 1485;
}
else
{
	vR[i] = 1484;
}

}

}
else
{
	if(input[i]>(1481+offset0))
{
	if(input[i]==(1483+offset0))
{
	vR[i] = 1483;
}
else
{
	vR[i] = 1482;
}

}
else
{
	if(input[i]==(1481+offset0))
{
	vR[i] = 1481;
}
else
{
	vR[i] = 1480;
}

}

}

}
else
{
	if(input[i]>(1475+offset0))
{
	if(input[i]>(1477+offset0))
{
	if(input[i]==(1479+offset0))
{
	vR[i] = 1479;
}
else
{
	vR[i] = 1478;
}

}
else
{
	if(input[i]==(1477+offset0))
{
	vR[i] = 1477;
}
else
{
	vR[i] = 1476;
}

}

}
else
{
	if(input[i]>(1473+offset0))
{
	if(input[i]==(1475+offset0))
{
	vR[i] = 1475;
}
else
{
	vR[i] = 1474;
}

}
else
{
	if(input[i]==(1473+offset0))
{
	vR[i] = 1473;
}
else
{
	vR[i] = 1472;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1439+offset0))
{
	if(input[i]>(1455+offset0))
{
	if(input[i]>(1463+offset0))
{
	if(input[i]>(1467+offset0))
{
	if(input[i]>(1469+offset0))
{
	if(input[i]==(1471+offset0))
{
	vR[i] = 1471;
}
else
{
	vR[i] = 1470;
}

}
else
{
	if(input[i]==(1469+offset0))
{
	vR[i] = 1469;
}
else
{
	vR[i] = 1468;
}

}

}
else
{
	if(input[i]>(1465+offset0))
{
	if(input[i]==(1467+offset0))
{
	vR[i] = 1467;
}
else
{
	vR[i] = 1466;
}

}
else
{
	if(input[i]==(1465+offset0))
{
	vR[i] = 1465;
}
else
{
	vR[i] = 1464;
}

}

}

}
else
{
	if(input[i]>(1459+offset0))
{
	if(input[i]>(1461+offset0))
{
	if(input[i]==(1463+offset0))
{
	vR[i] = 1463;
}
else
{
	vR[i] = 1462;
}

}
else
{
	if(input[i]==(1461+offset0))
{
	vR[i] = 1461;
}
else
{
	vR[i] = 1460;
}

}

}
else
{
	if(input[i]>(1457+offset0))
{
	if(input[i]==(1459+offset0))
{
	vR[i] = 1459;
}
else
{
	vR[i] = 1458;
}

}
else
{
	if(input[i]==(1457+offset0))
{
	vR[i] = 1457;
}
else
{
	vR[i] = 1456;
}

}

}

}

}
else
{
	if(input[i]>(1447+offset0))
{
	if(input[i]>(1451+offset0))
{
	if(input[i]>(1453+offset0))
{
	if(input[i]==(1455+offset0))
{
	vR[i] = 1455;
}
else
{
	vR[i] = 1454;
}

}
else
{
	if(input[i]==(1453+offset0))
{
	vR[i] = 1453;
}
else
{
	vR[i] = 1452;
}

}

}
else
{
	if(input[i]>(1449+offset0))
{
	if(input[i]==(1451+offset0))
{
	vR[i] = 1451;
}
else
{
	vR[i] = 1450;
}

}
else
{
	if(input[i]==(1449+offset0))
{
	vR[i] = 1449;
}
else
{
	vR[i] = 1448;
}

}

}

}
else
{
	if(input[i]>(1443+offset0))
{
	if(input[i]>(1445+offset0))
{
	if(input[i]==(1447+offset0))
{
	vR[i] = 1447;
}
else
{
	vR[i] = 1446;
}

}
else
{
	if(input[i]==(1445+offset0))
{
	vR[i] = 1445;
}
else
{
	vR[i] = 1444;
}

}

}
else
{
	if(input[i]>(1441+offset0))
{
	if(input[i]==(1443+offset0))
{
	vR[i] = 1443;
}
else
{
	vR[i] = 1442;
}

}
else
{
	if(input[i]==(1441+offset0))
{
	vR[i] = 1441;
}
else
{
	vR[i] = 1440;
}

}

}

}

}

}
else
{
	if(input[i]>(1423+offset0))
{
	if(input[i]>(1431+offset0))
{
	if(input[i]>(1435+offset0))
{
	if(input[i]>(1437+offset0))
{
	if(input[i]==(1439+offset0))
{
	vR[i] = 1439;
}
else
{
	vR[i] = 1438;
}

}
else
{
	if(input[i]==(1437+offset0))
{
	vR[i] = 1437;
}
else
{
	vR[i] = 1436;
}

}

}
else
{
	if(input[i]>(1433+offset0))
{
	if(input[i]==(1435+offset0))
{
	vR[i] = 1435;
}
else
{
	vR[i] = 1434;
}

}
else
{
	if(input[i]==(1433+offset0))
{
	vR[i] = 1433;
}
else
{
	vR[i] = 1432;
}

}

}

}
else
{
	if(input[i]>(1427+offset0))
{
	if(input[i]>(1429+offset0))
{
	if(input[i]==(1431+offset0))
{
	vR[i] = 1431;
}
else
{
	vR[i] = 1430;
}

}
else
{
	if(input[i]==(1429+offset0))
{
	vR[i] = 1429;
}
else
{
	vR[i] = 1428;
}

}

}
else
{
	if(input[i]>(1425+offset0))
{
	if(input[i]==(1427+offset0))
{
	vR[i] = 1427;
}
else
{
	vR[i] = 1426;
}

}
else
{
	if(input[i]==(1425+offset0))
{
	vR[i] = 1425;
}
else
{
	vR[i] = 1424;
}

}

}

}

}
else
{
	if(input[i]>(1415+offset0))
{
	if(input[i]>(1419+offset0))
{
	if(input[i]>(1421+offset0))
{
	if(input[i]==(1423+offset0))
{
	vR[i] = 1423;
}
else
{
	vR[i] = 1422;
}

}
else
{
	if(input[i]==(1421+offset0))
{
	vR[i] = 1421;
}
else
{
	vR[i] = 1420;
}

}

}
else
{
	if(input[i]>(1417+offset0))
{
	if(input[i]==(1419+offset0))
{
	vR[i] = 1419;
}
else
{
	vR[i] = 1418;
}

}
else
{
	if(input[i]==(1417+offset0))
{
	vR[i] = 1417;
}
else
{
	vR[i] = 1416;
}

}

}

}
else
{
	if(input[i]>(1411+offset0))
{
	if(input[i]>(1413+offset0))
{
	if(input[i]==(1415+offset0))
{
	vR[i] = 1415;
}
else
{
	vR[i] = 1414;
}

}
else
{
	if(input[i]==(1413+offset0))
{
	vR[i] = 1413;
}
else
{
	vR[i] = 1412;
}

}

}
else
{
	if(input[i]>(1409+offset0))
{
	if(input[i]==(1411+offset0))
{
	vR[i] = 1411;
}
else
{
	vR[i] = 1410;
}

}
else
{
	if(input[i]==(1409+offset0))
{
	vR[i] = 1409;
}
else
{
	vR[i] = 1408;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(1343+offset0))
{
	if(input[i]>(1375+offset0))
{
	if(input[i]>(1391+offset0))
{
	if(input[i]>(1399+offset0))
{
	if(input[i]>(1403+offset0))
{
	if(input[i]>(1405+offset0))
{
	if(input[i]==(1407+offset0))
{
	vR[i] = 1407;
}
else
{
	vR[i] = 1406;
}

}
else
{
	if(input[i]==(1405+offset0))
{
	vR[i] = 1405;
}
else
{
	vR[i] = 1404;
}

}

}
else
{
	if(input[i]>(1401+offset0))
{
	if(input[i]==(1403+offset0))
{
	vR[i] = 1403;
}
else
{
	vR[i] = 1402;
}

}
else
{
	if(input[i]==(1401+offset0))
{
	vR[i] = 1401;
}
else
{
	vR[i] = 1400;
}

}

}

}
else
{
	if(input[i]>(1395+offset0))
{
	if(input[i]>(1397+offset0))
{
	if(input[i]==(1399+offset0))
{
	vR[i] = 1399;
}
else
{
	vR[i] = 1398;
}

}
else
{
	if(input[i]==(1397+offset0))
{
	vR[i] = 1397;
}
else
{
	vR[i] = 1396;
}

}

}
else
{
	if(input[i]>(1393+offset0))
{
	if(input[i]==(1395+offset0))
{
	vR[i] = 1395;
}
else
{
	vR[i] = 1394;
}

}
else
{
	if(input[i]==(1393+offset0))
{
	vR[i] = 1393;
}
else
{
	vR[i] = 1392;
}

}

}

}

}
else
{
	if(input[i]>(1383+offset0))
{
	if(input[i]>(1387+offset0))
{
	if(input[i]>(1389+offset0))
{
	if(input[i]==(1391+offset0))
{
	vR[i] = 1391;
}
else
{
	vR[i] = 1390;
}

}
else
{
	if(input[i]==(1389+offset0))
{
	vR[i] = 1389;
}
else
{
	vR[i] = 1388;
}

}

}
else
{
	if(input[i]>(1385+offset0))
{
	if(input[i]==(1387+offset0))
{
	vR[i] = 1387;
}
else
{
	vR[i] = 1386;
}

}
else
{
	if(input[i]==(1385+offset0))
{
	vR[i] = 1385;
}
else
{
	vR[i] = 1384;
}

}

}

}
else
{
	if(input[i]>(1379+offset0))
{
	if(input[i]>(1381+offset0))
{
	if(input[i]==(1383+offset0))
{
	vR[i] = 1383;
}
else
{
	vR[i] = 1382;
}

}
else
{
	if(input[i]==(1381+offset0))
{
	vR[i] = 1381;
}
else
{
	vR[i] = 1380;
}

}

}
else
{
	if(input[i]>(1377+offset0))
{
	if(input[i]==(1379+offset0))
{
	vR[i] = 1379;
}
else
{
	vR[i] = 1378;
}

}
else
{
	if(input[i]==(1377+offset0))
{
	vR[i] = 1377;
}
else
{
	vR[i] = 1376;
}

}

}

}

}

}
else
{
	if(input[i]>(1359+offset0))
{
	if(input[i]>(1367+offset0))
{
	if(input[i]>(1371+offset0))
{
	if(input[i]>(1373+offset0))
{
	if(input[i]==(1375+offset0))
{
	vR[i] = 1375;
}
else
{
	vR[i] = 1374;
}

}
else
{
	if(input[i]==(1373+offset0))
{
	vR[i] = 1373;
}
else
{
	vR[i] = 1372;
}

}

}
else
{
	if(input[i]>(1369+offset0))
{
	if(input[i]==(1371+offset0))
{
	vR[i] = 1371;
}
else
{
	vR[i] = 1370;
}

}
else
{
	if(input[i]==(1369+offset0))
{
	vR[i] = 1369;
}
else
{
	vR[i] = 1368;
}

}

}

}
else
{
	if(input[i]>(1363+offset0))
{
	if(input[i]>(1365+offset0))
{
	if(input[i]==(1367+offset0))
{
	vR[i] = 1367;
}
else
{
	vR[i] = 1366;
}

}
else
{
	if(input[i]==(1365+offset0))
{
	vR[i] = 1365;
}
else
{
	vR[i] = 1364;
}

}

}
else
{
	if(input[i]>(1361+offset0))
{
	if(input[i]==(1363+offset0))
{
	vR[i] = 1363;
}
else
{
	vR[i] = 1362;
}

}
else
{
	if(input[i]==(1361+offset0))
{
	vR[i] = 1361;
}
else
{
	vR[i] = 1360;
}

}

}

}

}
else
{
	if(input[i]>(1351+offset0))
{
	if(input[i]>(1355+offset0))
{
	if(input[i]>(1357+offset0))
{
	if(input[i]==(1359+offset0))
{
	vR[i] = 1359;
}
else
{
	vR[i] = 1358;
}

}
else
{
	if(input[i]==(1357+offset0))
{
	vR[i] = 1357;
}
else
{
	vR[i] = 1356;
}

}

}
else
{
	if(input[i]>(1353+offset0))
{
	if(input[i]==(1355+offset0))
{
	vR[i] = 1355;
}
else
{
	vR[i] = 1354;
}

}
else
{
	if(input[i]==(1353+offset0))
{
	vR[i] = 1353;
}
else
{
	vR[i] = 1352;
}

}

}

}
else
{
	if(input[i]>(1347+offset0))
{
	if(input[i]>(1349+offset0))
{
	if(input[i]==(1351+offset0))
{
	vR[i] = 1351;
}
else
{
	vR[i] = 1350;
}

}
else
{
	if(input[i]==(1349+offset0))
{
	vR[i] = 1349;
}
else
{
	vR[i] = 1348;
}

}

}
else
{
	if(input[i]>(1345+offset0))
{
	if(input[i]==(1347+offset0))
{
	vR[i] = 1347;
}
else
{
	vR[i] = 1346;
}

}
else
{
	if(input[i]==(1345+offset0))
{
	vR[i] = 1345;
}
else
{
	vR[i] = 1344;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1311+offset0))
{
	if(input[i]>(1327+offset0))
{
	if(input[i]>(1335+offset0))
{
	if(input[i]>(1339+offset0))
{
	if(input[i]>(1341+offset0))
{
	if(input[i]==(1343+offset0))
{
	vR[i] = 1343;
}
else
{
	vR[i] = 1342;
}

}
else
{
	if(input[i]==(1341+offset0))
{
	vR[i] = 1341;
}
else
{
	vR[i] = 1340;
}

}

}
else
{
	if(input[i]>(1337+offset0))
{
	if(input[i]==(1339+offset0))
{
	vR[i] = 1339;
}
else
{
	vR[i] = 1338;
}

}
else
{
	if(input[i]==(1337+offset0))
{
	vR[i] = 1337;
}
else
{
	vR[i] = 1336;
}

}

}

}
else
{
	if(input[i]>(1331+offset0))
{
	if(input[i]>(1333+offset0))
{
	if(input[i]==(1335+offset0))
{
	vR[i] = 1335;
}
else
{
	vR[i] = 1334;
}

}
else
{
	if(input[i]==(1333+offset0))
{
	vR[i] = 1333;
}
else
{
	vR[i] = 1332;
}

}

}
else
{
	if(input[i]>(1329+offset0))
{
	if(input[i]==(1331+offset0))
{
	vR[i] = 1331;
}
else
{
	vR[i] = 1330;
}

}
else
{
	if(input[i]==(1329+offset0))
{
	vR[i] = 1329;
}
else
{
	vR[i] = 1328;
}

}

}

}

}
else
{
	if(input[i]>(1319+offset0))
{
	if(input[i]>(1323+offset0))
{
	if(input[i]>(1325+offset0))
{
	if(input[i]==(1327+offset0))
{
	vR[i] = 1327;
}
else
{
	vR[i] = 1326;
}

}
else
{
	if(input[i]==(1325+offset0))
{
	vR[i] = 1325;
}
else
{
	vR[i] = 1324;
}

}

}
else
{
	if(input[i]>(1321+offset0))
{
	if(input[i]==(1323+offset0))
{
	vR[i] = 1323;
}
else
{
	vR[i] = 1322;
}

}
else
{
	if(input[i]==(1321+offset0))
{
	vR[i] = 1321;
}
else
{
	vR[i] = 1320;
}

}

}

}
else
{
	if(input[i]>(1315+offset0))
{
	if(input[i]>(1317+offset0))
{
	if(input[i]==(1319+offset0))
{
	vR[i] = 1319;
}
else
{
	vR[i] = 1318;
}

}
else
{
	if(input[i]==(1317+offset0))
{
	vR[i] = 1317;
}
else
{
	vR[i] = 1316;
}

}

}
else
{
	if(input[i]>(1313+offset0))
{
	if(input[i]==(1315+offset0))
{
	vR[i] = 1315;
}
else
{
	vR[i] = 1314;
}

}
else
{
	if(input[i]==(1313+offset0))
{
	vR[i] = 1313;
}
else
{
	vR[i] = 1312;
}

}

}

}

}

}
else
{
	if(input[i]>(1295+offset0))
{
	if(input[i]>(1303+offset0))
{
	if(input[i]>(1307+offset0))
{
	if(input[i]>(1309+offset0))
{
	if(input[i]==(1311+offset0))
{
	vR[i] = 1311;
}
else
{
	vR[i] = 1310;
}

}
else
{
	if(input[i]==(1309+offset0))
{
	vR[i] = 1309;
}
else
{
	vR[i] = 1308;
}

}

}
else
{
	if(input[i]>(1305+offset0))
{
	if(input[i]==(1307+offset0))
{
	vR[i] = 1307;
}
else
{
	vR[i] = 1306;
}

}
else
{
	if(input[i]==(1305+offset0))
{
	vR[i] = 1305;
}
else
{
	vR[i] = 1304;
}

}

}

}
else
{
	if(input[i]>(1299+offset0))
{
	if(input[i]>(1301+offset0))
{
	if(input[i]==(1303+offset0))
{
	vR[i] = 1303;
}
else
{
	vR[i] = 1302;
}

}
else
{
	if(input[i]==(1301+offset0))
{
	vR[i] = 1301;
}
else
{
	vR[i] = 1300;
}

}

}
else
{
	if(input[i]>(1297+offset0))
{
	if(input[i]==(1299+offset0))
{
	vR[i] = 1299;
}
else
{
	vR[i] = 1298;
}

}
else
{
	if(input[i]==(1297+offset0))
{
	vR[i] = 1297;
}
else
{
	vR[i] = 1296;
}

}

}

}

}
else
{
	if(input[i]>(1287+offset0))
{
	if(input[i]>(1291+offset0))
{
	if(input[i]>(1293+offset0))
{
	if(input[i]==(1295+offset0))
{
	vR[i] = 1295;
}
else
{
	vR[i] = 1294;
}

}
else
{
	if(input[i]==(1293+offset0))
{
	vR[i] = 1293;
}
else
{
	vR[i] = 1292;
}

}

}
else
{
	if(input[i]>(1289+offset0))
{
	if(input[i]==(1291+offset0))
{
	vR[i] = 1291;
}
else
{
	vR[i] = 1290;
}

}
else
{
	if(input[i]==(1289+offset0))
{
	vR[i] = 1289;
}
else
{
	vR[i] = 1288;
}

}

}

}
else
{
	if(input[i]>(1283+offset0))
{
	if(input[i]>(1285+offset0))
{
	if(input[i]==(1287+offset0))
{
	vR[i] = 1287;
}
else
{
	vR[i] = 1286;
}

}
else
{
	if(input[i]==(1285+offset0))
{
	vR[i] = 1285;
}
else
{
	vR[i] = 1284;
}

}

}
else
{
	if(input[i]>(1281+offset0))
{
	if(input[i]==(1283+offset0))
{
	vR[i] = 1283;
}
else
{
	vR[i] = 1282;
}

}
else
{
	if(input[i]==(1281+offset0))
{
	vR[i] = 1281;
}
else
{
	vR[i] = 1280;
}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(1151+offset0))
{
	if(input[i]>(1215+offset0))
{
	if(input[i]>(1247+offset0))
{
	if(input[i]>(1263+offset0))
{
	if(input[i]>(1271+offset0))
{
	if(input[i]>(1275+offset0))
{
	if(input[i]>(1277+offset0))
{
	if(input[i]==(1279+offset0))
{
	vR[i] = 1279;
}
else
{
	vR[i] = 1278;
}

}
else
{
	if(input[i]==(1277+offset0))
{
	vR[i] = 1277;
}
else
{
	vR[i] = 1276;
}

}

}
else
{
	if(input[i]>(1273+offset0))
{
	if(input[i]==(1275+offset0))
{
	vR[i] = 1275;
}
else
{
	vR[i] = 1274;
}

}
else
{
	if(input[i]==(1273+offset0))
{
	vR[i] = 1273;
}
else
{
	vR[i] = 1272;
}

}

}

}
else
{
	if(input[i]>(1267+offset0))
{
	if(input[i]>(1269+offset0))
{
	if(input[i]==(1271+offset0))
{
	vR[i] = 1271;
}
else
{
	vR[i] = 1270;
}

}
else
{
	if(input[i]==(1269+offset0))
{
	vR[i] = 1269;
}
else
{
	vR[i] = 1268;
}

}

}
else
{
	if(input[i]>(1265+offset0))
{
	if(input[i]==(1267+offset0))
{
	vR[i] = 1267;
}
else
{
	vR[i] = 1266;
}

}
else
{
	if(input[i]==(1265+offset0))
{
	vR[i] = 1265;
}
else
{
	vR[i] = 1264;
}

}

}

}

}
else
{
	if(input[i]>(1255+offset0))
{
	if(input[i]>(1259+offset0))
{
	if(input[i]>(1261+offset0))
{
	if(input[i]==(1263+offset0))
{
	vR[i] = 1263;
}
else
{
	vR[i] = 1262;
}

}
else
{
	if(input[i]==(1261+offset0))
{
	vR[i] = 1261;
}
else
{
	vR[i] = 1260;
}

}

}
else
{
	if(input[i]>(1257+offset0))
{
	if(input[i]==(1259+offset0))
{
	vR[i] = 1259;
}
else
{
	vR[i] = 1258;
}

}
else
{
	if(input[i]==(1257+offset0))
{
	vR[i] = 1257;
}
else
{
	vR[i] = 1256;
}

}

}

}
else
{
	if(input[i]>(1251+offset0))
{
	if(input[i]>(1253+offset0))
{
	if(input[i]==(1255+offset0))
{
	vR[i] = 1255;
}
else
{
	vR[i] = 1254;
}

}
else
{
	if(input[i]==(1253+offset0))
{
	vR[i] = 1253;
}
else
{
	vR[i] = 1252;
}

}

}
else
{
	if(input[i]>(1249+offset0))
{
	if(input[i]==(1251+offset0))
{
	vR[i] = 1251;
}
else
{
	vR[i] = 1250;
}

}
else
{
	if(input[i]==(1249+offset0))
{
	vR[i] = 1249;
}
else
{
	vR[i] = 1248;
}

}

}

}

}

}
else
{
	if(input[i]>(1231+offset0))
{
	if(input[i]>(1239+offset0))
{
	if(input[i]>(1243+offset0))
{
	if(input[i]>(1245+offset0))
{
	if(input[i]==(1247+offset0))
{
	vR[i] = 1247;
}
else
{
	vR[i] = 1246;
}

}
else
{
	if(input[i]==(1245+offset0))
{
	vR[i] = 1245;
}
else
{
	vR[i] = 1244;
}

}

}
else
{
	if(input[i]>(1241+offset0))
{
	if(input[i]==(1243+offset0))
{
	vR[i] = 1243;
}
else
{
	vR[i] = 1242;
}

}
else
{
	if(input[i]==(1241+offset0))
{
	vR[i] = 1241;
}
else
{
	vR[i] = 1240;
}

}

}

}
else
{
	if(input[i]>(1235+offset0))
{
	if(input[i]>(1237+offset0))
{
	if(input[i]==(1239+offset0))
{
	vR[i] = 1239;
}
else
{
	vR[i] = 1238;
}

}
else
{
	if(input[i]==(1237+offset0))
{
	vR[i] = 1237;
}
else
{
	vR[i] = 1236;
}

}

}
else
{
	if(input[i]>(1233+offset0))
{
	if(input[i]==(1235+offset0))
{
	vR[i] = 1235;
}
else
{
	vR[i] = 1234;
}

}
else
{
	if(input[i]==(1233+offset0))
{
	vR[i] = 1233;
}
else
{
	vR[i] = 1232;
}

}

}

}

}
else
{
	if(input[i]>(1223+offset0))
{
	if(input[i]>(1227+offset0))
{
	if(input[i]>(1229+offset0))
{
	if(input[i]==(1231+offset0))
{
	vR[i] = 1231;
}
else
{
	vR[i] = 1230;
}

}
else
{
	if(input[i]==(1229+offset0))
{
	vR[i] = 1229;
}
else
{
	vR[i] = 1228;
}

}

}
else
{
	if(input[i]>(1225+offset0))
{
	if(input[i]==(1227+offset0))
{
	vR[i] = 1227;
}
else
{
	vR[i] = 1226;
}

}
else
{
	if(input[i]==(1225+offset0))
{
	vR[i] = 1225;
}
else
{
	vR[i] = 1224;
}

}

}

}
else
{
	if(input[i]>(1219+offset0))
{
	if(input[i]>(1221+offset0))
{
	if(input[i]==(1223+offset0))
{
	vR[i] = 1223;
}
else
{
	vR[i] = 1222;
}

}
else
{
	if(input[i]==(1221+offset0))
{
	vR[i] = 1221;
}
else
{
	vR[i] = 1220;
}

}

}
else
{
	if(input[i]>(1217+offset0))
{
	if(input[i]==(1219+offset0))
{
	vR[i] = 1219;
}
else
{
	vR[i] = 1218;
}

}
else
{
	if(input[i]==(1217+offset0))
{
	vR[i] = 1217;
}
else
{
	vR[i] = 1216;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1183+offset0))
{
	if(input[i]>(1199+offset0))
{
	if(input[i]>(1207+offset0))
{
	if(input[i]>(1211+offset0))
{
	if(input[i]>(1213+offset0))
{
	if(input[i]==(1215+offset0))
{
	vR[i] = 1215;
}
else
{
	vR[i] = 1214;
}

}
else
{
	if(input[i]==(1213+offset0))
{
	vR[i] = 1213;
}
else
{
	vR[i] = 1212;
}

}

}
else
{
	if(input[i]>(1209+offset0))
{
	if(input[i]==(1211+offset0))
{
	vR[i] = 1211;
}
else
{
	vR[i] = 1210;
}

}
else
{
	if(input[i]==(1209+offset0))
{
	vR[i] = 1209;
}
else
{
	vR[i] = 1208;
}

}

}

}
else
{
	if(input[i]>(1203+offset0))
{
	if(input[i]>(1205+offset0))
{
	if(input[i]==(1207+offset0))
{
	vR[i] = 1207;
}
else
{
	vR[i] = 1206;
}

}
else
{
	if(input[i]==(1205+offset0))
{
	vR[i] = 1205;
}
else
{
	vR[i] = 1204;
}

}

}
else
{
	if(input[i]>(1201+offset0))
{
	if(input[i]==(1203+offset0))
{
	vR[i] = 1203;
}
else
{
	vR[i] = 1202;
}

}
else
{
	if(input[i]==(1201+offset0))
{
	vR[i] = 1201;
}
else
{
	vR[i] = 1200;
}

}

}

}

}
else
{
	if(input[i]>(1191+offset0))
{
	if(input[i]>(1195+offset0))
{
	if(input[i]>(1197+offset0))
{
	if(input[i]==(1199+offset0))
{
	vR[i] = 1199;
}
else
{
	vR[i] = 1198;
}

}
else
{
	if(input[i]==(1197+offset0))
{
	vR[i] = 1197;
}
else
{
	vR[i] = 1196;
}

}

}
else
{
	if(input[i]>(1193+offset0))
{
	if(input[i]==(1195+offset0))
{
	vR[i] = 1195;
}
else
{
	vR[i] = 1194;
}

}
else
{
	if(input[i]==(1193+offset0))
{
	vR[i] = 1193;
}
else
{
	vR[i] = 1192;
}

}

}

}
else
{
	if(input[i]>(1187+offset0))
{
	if(input[i]>(1189+offset0))
{
	if(input[i]==(1191+offset0))
{
	vR[i] = 1191;
}
else
{
	vR[i] = 1190;
}

}
else
{
	if(input[i]==(1189+offset0))
{
	vR[i] = 1189;
}
else
{
	vR[i] = 1188;
}

}

}
else
{
	if(input[i]>(1185+offset0))
{
	if(input[i]==(1187+offset0))
{
	vR[i] = 1187;
}
else
{
	vR[i] = 1186;
}

}
else
{
	if(input[i]==(1185+offset0))
{
	vR[i] = 1185;
}
else
{
	vR[i] = 1184;
}

}

}

}

}

}
else
{
	if(input[i]>(1167+offset0))
{
	if(input[i]>(1175+offset0))
{
	if(input[i]>(1179+offset0))
{
	if(input[i]>(1181+offset0))
{
	if(input[i]==(1183+offset0))
{
	vR[i] = 1183;
}
else
{
	vR[i] = 1182;
}

}
else
{
	if(input[i]==(1181+offset0))
{
	vR[i] = 1181;
}
else
{
	vR[i] = 1180;
}

}

}
else
{
	if(input[i]>(1177+offset0))
{
	if(input[i]==(1179+offset0))
{
	vR[i] = 1179;
}
else
{
	vR[i] = 1178;
}

}
else
{
	if(input[i]==(1177+offset0))
{
	vR[i] = 1177;
}
else
{
	vR[i] = 1176;
}

}

}

}
else
{
	if(input[i]>(1171+offset0))
{
	if(input[i]>(1173+offset0))
{
	if(input[i]==(1175+offset0))
{
	vR[i] = 1175;
}
else
{
	vR[i] = 1174;
}

}
else
{
	if(input[i]==(1173+offset0))
{
	vR[i] = 1173;
}
else
{
	vR[i] = 1172;
}

}

}
else
{
	if(input[i]>(1169+offset0))
{
	if(input[i]==(1171+offset0))
{
	vR[i] = 1171;
}
else
{
	vR[i] = 1170;
}

}
else
{
	if(input[i]==(1169+offset0))
{
	vR[i] = 1169;
}
else
{
	vR[i] = 1168;
}

}

}

}

}
else
{
	if(input[i]>(1159+offset0))
{
	if(input[i]>(1163+offset0))
{
	if(input[i]>(1165+offset0))
{
	if(input[i]==(1167+offset0))
{
	vR[i] = 1167;
}
else
{
	vR[i] = 1166;
}

}
else
{
	if(input[i]==(1165+offset0))
{
	vR[i] = 1165;
}
else
{
	vR[i] = 1164;
}

}

}
else
{
	if(input[i]>(1161+offset0))
{
	if(input[i]==(1163+offset0))
{
	vR[i] = 1163;
}
else
{
	vR[i] = 1162;
}

}
else
{
	if(input[i]==(1161+offset0))
{
	vR[i] = 1161;
}
else
{
	vR[i] = 1160;
}

}

}

}
else
{
	if(input[i]>(1155+offset0))
{
	if(input[i]>(1157+offset0))
{
	if(input[i]==(1159+offset0))
{
	vR[i] = 1159;
}
else
{
	vR[i] = 1158;
}

}
else
{
	if(input[i]==(1157+offset0))
{
	vR[i] = 1157;
}
else
{
	vR[i] = 1156;
}

}

}
else
{
	if(input[i]>(1153+offset0))
{
	if(input[i]==(1155+offset0))
{
	vR[i] = 1155;
}
else
{
	vR[i] = 1154;
}

}
else
{
	if(input[i]==(1153+offset0))
{
	vR[i] = 1153;
}
else
{
	vR[i] = 1152;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(1087+offset0))
{
	if(input[i]>(1119+offset0))
{
	if(input[i]>(1135+offset0))
{
	if(input[i]>(1143+offset0))
{
	if(input[i]>(1147+offset0))
{
	if(input[i]>(1149+offset0))
{
	if(input[i]==(1151+offset0))
{
	vR[i] = 1151;
}
else
{
	vR[i] = 1150;
}

}
else
{
	if(input[i]==(1149+offset0))
{
	vR[i] = 1149;
}
else
{
	vR[i] = 1148;
}

}

}
else
{
	if(input[i]>(1145+offset0))
{
	if(input[i]==(1147+offset0))
{
	vR[i] = 1147;
}
else
{
	vR[i] = 1146;
}

}
else
{
	if(input[i]==(1145+offset0))
{
	vR[i] = 1145;
}
else
{
	vR[i] = 1144;
}

}

}

}
else
{
	if(input[i]>(1139+offset0))
{
	if(input[i]>(1141+offset0))
{
	if(input[i]==(1143+offset0))
{
	vR[i] = 1143;
}
else
{
	vR[i] = 1142;
}

}
else
{
	if(input[i]==(1141+offset0))
{
	vR[i] = 1141;
}
else
{
	vR[i] = 1140;
}

}

}
else
{
	if(input[i]>(1137+offset0))
{
	if(input[i]==(1139+offset0))
{
	vR[i] = 1139;
}
else
{
	vR[i] = 1138;
}

}
else
{
	if(input[i]==(1137+offset0))
{
	vR[i] = 1137;
}
else
{
	vR[i] = 1136;
}

}

}

}

}
else
{
	if(input[i]>(1127+offset0))
{
	if(input[i]>(1131+offset0))
{
	if(input[i]>(1133+offset0))
{
	if(input[i]==(1135+offset0))
{
	vR[i] = 1135;
}
else
{
	vR[i] = 1134;
}

}
else
{
	if(input[i]==(1133+offset0))
{
	vR[i] = 1133;
}
else
{
	vR[i] = 1132;
}

}

}
else
{
	if(input[i]>(1129+offset0))
{
	if(input[i]==(1131+offset0))
{
	vR[i] = 1131;
}
else
{
	vR[i] = 1130;
}

}
else
{
	if(input[i]==(1129+offset0))
{
	vR[i] = 1129;
}
else
{
	vR[i] = 1128;
}

}

}

}
else
{
	if(input[i]>(1123+offset0))
{
	if(input[i]>(1125+offset0))
{
	if(input[i]==(1127+offset0))
{
	vR[i] = 1127;
}
else
{
	vR[i] = 1126;
}

}
else
{
	if(input[i]==(1125+offset0))
{
	vR[i] = 1125;
}
else
{
	vR[i] = 1124;
}

}

}
else
{
	if(input[i]>(1121+offset0))
{
	if(input[i]==(1123+offset0))
{
	vR[i] = 1123;
}
else
{
	vR[i] = 1122;
}

}
else
{
	if(input[i]==(1121+offset0))
{
	vR[i] = 1121;
}
else
{
	vR[i] = 1120;
}

}

}

}

}

}
else
{
	if(input[i]>(1103+offset0))
{
	if(input[i]>(1111+offset0))
{
	if(input[i]>(1115+offset0))
{
	if(input[i]>(1117+offset0))
{
	if(input[i]==(1119+offset0))
{
	vR[i] = 1119;
}
else
{
	vR[i] = 1118;
}

}
else
{
	if(input[i]==(1117+offset0))
{
	vR[i] = 1117;
}
else
{
	vR[i] = 1116;
}

}

}
else
{
	if(input[i]>(1113+offset0))
{
	if(input[i]==(1115+offset0))
{
	vR[i] = 1115;
}
else
{
	vR[i] = 1114;
}

}
else
{
	if(input[i]==(1113+offset0))
{
	vR[i] = 1113;
}
else
{
	vR[i] = 1112;
}

}

}

}
else
{
	if(input[i]>(1107+offset0))
{
	if(input[i]>(1109+offset0))
{
	if(input[i]==(1111+offset0))
{
	vR[i] = 1111;
}
else
{
	vR[i] = 1110;
}

}
else
{
	if(input[i]==(1109+offset0))
{
	vR[i] = 1109;
}
else
{
	vR[i] = 1108;
}

}

}
else
{
	if(input[i]>(1105+offset0))
{
	if(input[i]==(1107+offset0))
{
	vR[i] = 1107;
}
else
{
	vR[i] = 1106;
}

}
else
{
	if(input[i]==(1105+offset0))
{
	vR[i] = 1105;
}
else
{
	vR[i] = 1104;
}

}

}

}

}
else
{
	if(input[i]>(1095+offset0))
{
	if(input[i]>(1099+offset0))
{
	if(input[i]>(1101+offset0))
{
	if(input[i]==(1103+offset0))
{
	vR[i] = 1103;
}
else
{
	vR[i] = 1102;
}

}
else
{
	if(input[i]==(1101+offset0))
{
	vR[i] = 1101;
}
else
{
	vR[i] = 1100;
}

}

}
else
{
	if(input[i]>(1097+offset0))
{
	if(input[i]==(1099+offset0))
{
	vR[i] = 1099;
}
else
{
	vR[i] = 1098;
}

}
else
{
	if(input[i]==(1097+offset0))
{
	vR[i] = 1097;
}
else
{
	vR[i] = 1096;
}

}

}

}
else
{
	if(input[i]>(1091+offset0))
{
	if(input[i]>(1093+offset0))
{
	if(input[i]==(1095+offset0))
{
	vR[i] = 1095;
}
else
{
	vR[i] = 1094;
}

}
else
{
	if(input[i]==(1093+offset0))
{
	vR[i] = 1093;
}
else
{
	vR[i] = 1092;
}

}

}
else
{
	if(input[i]>(1089+offset0))
{
	if(input[i]==(1091+offset0))
{
	vR[i] = 1091;
}
else
{
	vR[i] = 1090;
}

}
else
{
	if(input[i]==(1089+offset0))
{
	vR[i] = 1089;
}
else
{
	vR[i] = 1088;
}

}

}

}

}

}

}
else
{
	if(input[i]>(1055+offset0))
{
	if(input[i]>(1071+offset0))
{
	if(input[i]>(1079+offset0))
{
	if(input[i]>(1083+offset0))
{
	if(input[i]>(1085+offset0))
{
	if(input[i]==(1087+offset0))
{
	vR[i] = 1087;
}
else
{
	vR[i] = 1086;
}

}
else
{
	if(input[i]==(1085+offset0))
{
	vR[i] = 1085;
}
else
{
	vR[i] = 1084;
}

}

}
else
{
	if(input[i]>(1081+offset0))
{
	if(input[i]==(1083+offset0))
{
	vR[i] = 1083;
}
else
{
	vR[i] = 1082;
}

}
else
{
	if(input[i]==(1081+offset0))
{
	vR[i] = 1081;
}
else
{
	vR[i] = 1080;
}

}

}

}
else
{
	if(input[i]>(1075+offset0))
{
	if(input[i]>(1077+offset0))
{
	if(input[i]==(1079+offset0))
{
	vR[i] = 1079;
}
else
{
	vR[i] = 1078;
}

}
else
{
	if(input[i]==(1077+offset0))
{
	vR[i] = 1077;
}
else
{
	vR[i] = 1076;
}

}

}
else
{
	if(input[i]>(1073+offset0))
{
	if(input[i]==(1075+offset0))
{
	vR[i] = 1075;
}
else
{
	vR[i] = 1074;
}

}
else
{
	if(input[i]==(1073+offset0))
{
	vR[i] = 1073;
}
else
{
	vR[i] = 1072;
}

}

}

}

}
else
{
	if(input[i]>(1063+offset0))
{
	if(input[i]>(1067+offset0))
{
	if(input[i]>(1069+offset0))
{
	if(input[i]==(1071+offset0))
{
	vR[i] = 1071;
}
else
{
	vR[i] = 1070;
}

}
else
{
	if(input[i]==(1069+offset0))
{
	vR[i] = 1069;
}
else
{
	vR[i] = 1068;
}

}

}
else
{
	if(input[i]>(1065+offset0))
{
	if(input[i]==(1067+offset0))
{
	vR[i] = 1067;
}
else
{
	vR[i] = 1066;
}

}
else
{
	if(input[i]==(1065+offset0))
{
	vR[i] = 1065;
}
else
{
	vR[i] = 1064;
}

}

}

}
else
{
	if(input[i]>(1059+offset0))
{
	if(input[i]>(1061+offset0))
{
	if(input[i]==(1063+offset0))
{
	vR[i] = 1063;
}
else
{
	vR[i] = 1062;
}

}
else
{
	if(input[i]==(1061+offset0))
{
	vR[i] = 1061;
}
else
{
	vR[i] = 1060;
}

}

}
else
{
	if(input[i]>(1057+offset0))
{
	if(input[i]==(1059+offset0))
{
	vR[i] = 1059;
}
else
{
	vR[i] = 1058;
}

}
else
{
	if(input[i]==(1057+offset0))
{
	vR[i] = 1057;
}
else
{
	vR[i] = 1056;
}

}

}

}

}

}
else
{
	if(input[i]>(1039+offset0))
{
	if(input[i]>(1047+offset0))
{
	if(input[i]>(1051+offset0))
{
	if(input[i]>(1053+offset0))
{
	if(input[i]==(1055+offset0))
{
	vR[i] = 1055;
}
else
{
	vR[i] = 1054;
}

}
else
{
	if(input[i]==(1053+offset0))
{
	vR[i] = 1053;
}
else
{
	vR[i] = 1052;
}

}

}
else
{
	if(input[i]>(1049+offset0))
{
	if(input[i]==(1051+offset0))
{
	vR[i] = 1051;
}
else
{
	vR[i] = 1050;
}

}
else
{
	if(input[i]==(1049+offset0))
{
	vR[i] = 1049;
}
else
{
	vR[i] = 1048;
}

}

}

}
else
{
	if(input[i]>(1043+offset0))
{
	if(input[i]>(1045+offset0))
{
	if(input[i]==(1047+offset0))
{
	vR[i] = 1047;
}
else
{
	vR[i] = 1046;
}

}
else
{
	if(input[i]==(1045+offset0))
{
	vR[i] = 1045;
}
else
{
	vR[i] = 1044;
}

}

}
else
{
	if(input[i]>(1041+offset0))
{
	if(input[i]==(1043+offset0))
{
	vR[i] = 1043;
}
else
{
	vR[i] = 1042;
}

}
else
{
	if(input[i]==(1041+offset0))
{
	vR[i] = 1041;
}
else
{
	vR[i] = 1040;
}

}

}

}

}
else
{
	if(input[i]>(1031+offset0))
{
	if(input[i]>(1035+offset0))
{
	if(input[i]>(1037+offset0))
{
	if(input[i]==(1039+offset0))
{
	vR[i] = 1039;
}
else
{
	vR[i] = 1038;
}

}
else
{
	if(input[i]==(1037+offset0))
{
	vR[i] = 1037;
}
else
{
	vR[i] = 1036;
}

}

}
else
{
	if(input[i]>(1033+offset0))
{
	if(input[i]==(1035+offset0))
{
	vR[i] = 1035;
}
else
{
	vR[i] = 1034;
}

}
else
{
	if(input[i]==(1033+offset0))
{
	vR[i] = 1033;
}
else
{
	vR[i] = 1032;
}

}

}

}
else
{
	if(input[i]>(1027+offset0))
{
	if(input[i]>(1029+offset0))
{
	if(input[i]==(1031+offset0))
{
	vR[i] = 1031;
}
else
{
	vR[i] = 1030;
}

}
else
{
	if(input[i]==(1029+offset0))
{
	vR[i] = 1029;
}
else
{
	vR[i] = 1028;
}

}

}
else
{
	if(input[i]>(1025+offset0))
{
	if(input[i]==(1027+offset0))
{
	vR[i] = 1027;
}
else
{
	vR[i] = 1026;
}

}
else
{
	if(input[i]==(1025+offset0))
{
	vR[i] = 1025;
}
else
{
	vR[i] = 1024;
}

}

}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(511+offset0))
{
	if(input[i]>(767+offset0))
{
	if(input[i]>(895+offset0))
{
	if(input[i]>(959+offset0))
{
	if(input[i]>(991+offset0))
{
	if(input[i]>(1007+offset0))
{
	if(input[i]>(1015+offset0))
{
	if(input[i]>(1019+offset0))
{
	if(input[i]>(1021+offset0))
{
	if(input[i]==(1023+offset0))
{
	vR[i] = 1023;
}
else
{
	vR[i] = 1022;
}

}
else
{
	if(input[i]==(1021+offset0))
{
	vR[i] = 1021;
}
else
{
	vR[i] = 1020;
}

}

}
else
{
	if(input[i]>(1017+offset0))
{
	if(input[i]==(1019+offset0))
{
	vR[i] = 1019;
}
else
{
	vR[i] = 1018;
}

}
else
{
	if(input[i]==(1017+offset0))
{
	vR[i] = 1017;
}
else
{
	vR[i] = 1016;
}

}

}

}
else
{
	if(input[i]>(1011+offset0))
{
	if(input[i]>(1013+offset0))
{
	if(input[i]==(1015+offset0))
{
	vR[i] = 1015;
}
else
{
	vR[i] = 1014;
}

}
else
{
	if(input[i]==(1013+offset0))
{
	vR[i] = 1013;
}
else
{
	vR[i] = 1012;
}

}

}
else
{
	if(input[i]>(1009+offset0))
{
	if(input[i]==(1011+offset0))
{
	vR[i] = 1011;
}
else
{
	vR[i] = 1010;
}

}
else
{
	if(input[i]==(1009+offset0))
{
	vR[i] = 1009;
}
else
{
	vR[i] = 1008;
}

}

}

}

}
else
{
	if(input[i]>(999+offset0))
{
	if(input[i]>(1003+offset0))
{
	if(input[i]>(1005+offset0))
{
	if(input[i]==(1007+offset0))
{
	vR[i] = 1007;
}
else
{
	vR[i] = 1006;
}

}
else
{
	if(input[i]==(1005+offset0))
{
	vR[i] = 1005;
}
else
{
	vR[i] = 1004;
}

}

}
else
{
	if(input[i]>(1001+offset0))
{
	if(input[i]==(1003+offset0))
{
	vR[i] = 1003;
}
else
{
	vR[i] = 1002;
}

}
else
{
	if(input[i]==(1001+offset0))
{
	vR[i] = 1001;
}
else
{
	vR[i] = 1000;
}

}

}

}
else
{
	if(input[i]>(995+offset0))
{
	if(input[i]>(997+offset0))
{
	if(input[i]==(999+offset0))
{
	vR[i] = 999;
}
else
{
	vR[i] = 998;
}

}
else
{
	if(input[i]==(997+offset0))
{
	vR[i] = 997;
}
else
{
	vR[i] = 996;
}

}

}
else
{
	if(input[i]>(993+offset0))
{
	if(input[i]==(995+offset0))
{
	vR[i] = 995;
}
else
{
	vR[i] = 994;
}

}
else
{
	if(input[i]==(993+offset0))
{
	vR[i] = 993;
}
else
{
	vR[i] = 992;
}

}

}

}

}

}
else
{
	if(input[i]>(975+offset0))
{
	if(input[i]>(983+offset0))
{
	if(input[i]>(987+offset0))
{
	if(input[i]>(989+offset0))
{
	if(input[i]==(991+offset0))
{
	vR[i] = 991;
}
else
{
	vR[i] = 990;
}

}
else
{
	if(input[i]==(989+offset0))
{
	vR[i] = 989;
}
else
{
	vR[i] = 988;
}

}

}
else
{
	if(input[i]>(985+offset0))
{
	if(input[i]==(987+offset0))
{
	vR[i] = 987;
}
else
{
	vR[i] = 986;
}

}
else
{
	if(input[i]==(985+offset0))
{
	vR[i] = 985;
}
else
{
	vR[i] = 984;
}

}

}

}
else
{
	if(input[i]>(979+offset0))
{
	if(input[i]>(981+offset0))
{
	if(input[i]==(983+offset0))
{
	vR[i] = 983;
}
else
{
	vR[i] = 982;
}

}
else
{
	if(input[i]==(981+offset0))
{
	vR[i] = 981;
}
else
{
	vR[i] = 980;
}

}

}
else
{
	if(input[i]>(977+offset0))
{
	if(input[i]==(979+offset0))
{
	vR[i] = 979;
}
else
{
	vR[i] = 978;
}

}
else
{
	if(input[i]==(977+offset0))
{
	vR[i] = 977;
}
else
{
	vR[i] = 976;
}

}

}

}

}
else
{
	if(input[i]>(967+offset0))
{
	if(input[i]>(971+offset0))
{
	if(input[i]>(973+offset0))
{
	if(input[i]==(975+offset0))
{
	vR[i] = 975;
}
else
{
	vR[i] = 974;
}

}
else
{
	if(input[i]==(973+offset0))
{
	vR[i] = 973;
}
else
{
	vR[i] = 972;
}

}

}
else
{
	if(input[i]>(969+offset0))
{
	if(input[i]==(971+offset0))
{
	vR[i] = 971;
}
else
{
	vR[i] = 970;
}

}
else
{
	if(input[i]==(969+offset0))
{
	vR[i] = 969;
}
else
{
	vR[i] = 968;
}

}

}

}
else
{
	if(input[i]>(963+offset0))
{
	if(input[i]>(965+offset0))
{
	if(input[i]==(967+offset0))
{
	vR[i] = 967;
}
else
{
	vR[i] = 966;
}

}
else
{
	if(input[i]==(965+offset0))
{
	vR[i] = 965;
}
else
{
	vR[i] = 964;
}

}

}
else
{
	if(input[i]>(961+offset0))
{
	if(input[i]==(963+offset0))
{
	vR[i] = 963;
}
else
{
	vR[i] = 962;
}

}
else
{
	if(input[i]==(961+offset0))
{
	vR[i] = 961;
}
else
{
	vR[i] = 960;
}

}

}

}

}

}

}
else
{
	if(input[i]>(927+offset0))
{
	if(input[i]>(943+offset0))
{
	if(input[i]>(951+offset0))
{
	if(input[i]>(955+offset0))
{
	if(input[i]>(957+offset0))
{
	if(input[i]==(959+offset0))
{
	vR[i] = 959;
}
else
{
	vR[i] = 958;
}

}
else
{
	if(input[i]==(957+offset0))
{
	vR[i] = 957;
}
else
{
	vR[i] = 956;
}

}

}
else
{
	if(input[i]>(953+offset0))
{
	if(input[i]==(955+offset0))
{
	vR[i] = 955;
}
else
{
	vR[i] = 954;
}

}
else
{
	if(input[i]==(953+offset0))
{
	vR[i] = 953;
}
else
{
	vR[i] = 952;
}

}

}

}
else
{
	if(input[i]>(947+offset0))
{
	if(input[i]>(949+offset0))
{
	if(input[i]==(951+offset0))
{
	vR[i] = 951;
}
else
{
	vR[i] = 950;
}

}
else
{
	if(input[i]==(949+offset0))
{
	vR[i] = 949;
}
else
{
	vR[i] = 948;
}

}

}
else
{
	if(input[i]>(945+offset0))
{
	if(input[i]==(947+offset0))
{
	vR[i] = 947;
}
else
{
	vR[i] = 946;
}

}
else
{
	if(input[i]==(945+offset0))
{
	vR[i] = 945;
}
else
{
	vR[i] = 944;
}

}

}

}

}
else
{
	if(input[i]>(935+offset0))
{
	if(input[i]>(939+offset0))
{
	if(input[i]>(941+offset0))
{
	if(input[i]==(943+offset0))
{
	vR[i] = 943;
}
else
{
	vR[i] = 942;
}

}
else
{
	if(input[i]==(941+offset0))
{
	vR[i] = 941;
}
else
{
	vR[i] = 940;
}

}

}
else
{
	if(input[i]>(937+offset0))
{
	if(input[i]==(939+offset0))
{
	vR[i] = 939;
}
else
{
	vR[i] = 938;
}

}
else
{
	if(input[i]==(937+offset0))
{
	vR[i] = 937;
}
else
{
	vR[i] = 936;
}

}

}

}
else
{
	if(input[i]>(931+offset0))
{
	if(input[i]>(933+offset0))
{
	if(input[i]==(935+offset0))
{
	vR[i] = 935;
}
else
{
	vR[i] = 934;
}

}
else
{
	if(input[i]==(933+offset0))
{
	vR[i] = 933;
}
else
{
	vR[i] = 932;
}

}

}
else
{
	if(input[i]>(929+offset0))
{
	if(input[i]==(931+offset0))
{
	vR[i] = 931;
}
else
{
	vR[i] = 930;
}

}
else
{
	if(input[i]==(929+offset0))
{
	vR[i] = 929;
}
else
{
	vR[i] = 928;
}

}

}

}

}

}
else
{
	if(input[i]>(911+offset0))
{
	if(input[i]>(919+offset0))
{
	if(input[i]>(923+offset0))
{
	if(input[i]>(925+offset0))
{
	if(input[i]==(927+offset0))
{
	vR[i] = 927;
}
else
{
	vR[i] = 926;
}

}
else
{
	if(input[i]==(925+offset0))
{
	vR[i] = 925;
}
else
{
	vR[i] = 924;
}

}

}
else
{
	if(input[i]>(921+offset0))
{
	if(input[i]==(923+offset0))
{
	vR[i] = 923;
}
else
{
	vR[i] = 922;
}

}
else
{
	if(input[i]==(921+offset0))
{
	vR[i] = 921;
}
else
{
	vR[i] = 920;
}

}

}

}
else
{
	if(input[i]>(915+offset0))
{
	if(input[i]>(917+offset0))
{
	if(input[i]==(919+offset0))
{
	vR[i] = 919;
}
else
{
	vR[i] = 918;
}

}
else
{
	if(input[i]==(917+offset0))
{
	vR[i] = 917;
}
else
{
	vR[i] = 916;
}

}

}
else
{
	if(input[i]>(913+offset0))
{
	if(input[i]==(915+offset0))
{
	vR[i] = 915;
}
else
{
	vR[i] = 914;
}

}
else
{
	if(input[i]==(913+offset0))
{
	vR[i] = 913;
}
else
{
	vR[i] = 912;
}

}

}

}

}
else
{
	if(input[i]>(903+offset0))
{
	if(input[i]>(907+offset0))
{
	if(input[i]>(909+offset0))
{
	if(input[i]==(911+offset0))
{
	vR[i] = 911;
}
else
{
	vR[i] = 910;
}

}
else
{
	if(input[i]==(909+offset0))
{
	vR[i] = 909;
}
else
{
	vR[i] = 908;
}

}

}
else
{
	if(input[i]>(905+offset0))
{
	if(input[i]==(907+offset0))
{
	vR[i] = 907;
}
else
{
	vR[i] = 906;
}

}
else
{
	if(input[i]==(905+offset0))
{
	vR[i] = 905;
}
else
{
	vR[i] = 904;
}

}

}

}
else
{
	if(input[i]>(899+offset0))
{
	if(input[i]>(901+offset0))
{
	if(input[i]==(903+offset0))
{
	vR[i] = 903;
}
else
{
	vR[i] = 902;
}

}
else
{
	if(input[i]==(901+offset0))
{
	vR[i] = 901;
}
else
{
	vR[i] = 900;
}

}

}
else
{
	if(input[i]>(897+offset0))
{
	if(input[i]==(899+offset0))
{
	vR[i] = 899;
}
else
{
	vR[i] = 898;
}

}
else
{
	if(input[i]==(897+offset0))
{
	vR[i] = 897;
}
else
{
	vR[i] = 896;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(831+offset0))
{
	if(input[i]>(863+offset0))
{
	if(input[i]>(879+offset0))
{
	if(input[i]>(887+offset0))
{
	if(input[i]>(891+offset0))
{
	if(input[i]>(893+offset0))
{
	if(input[i]==(895+offset0))
{
	vR[i] = 895;
}
else
{
	vR[i] = 894;
}

}
else
{
	if(input[i]==(893+offset0))
{
	vR[i] = 893;
}
else
{
	vR[i] = 892;
}

}

}
else
{
	if(input[i]>(889+offset0))
{
	if(input[i]==(891+offset0))
{
	vR[i] = 891;
}
else
{
	vR[i] = 890;
}

}
else
{
	if(input[i]==(889+offset0))
{
	vR[i] = 889;
}
else
{
	vR[i] = 888;
}

}

}

}
else
{
	if(input[i]>(883+offset0))
{
	if(input[i]>(885+offset0))
{
	if(input[i]==(887+offset0))
{
	vR[i] = 887;
}
else
{
	vR[i] = 886;
}

}
else
{
	if(input[i]==(885+offset0))
{
	vR[i] = 885;
}
else
{
	vR[i] = 884;
}

}

}
else
{
	if(input[i]>(881+offset0))
{
	if(input[i]==(883+offset0))
{
	vR[i] = 883;
}
else
{
	vR[i] = 882;
}

}
else
{
	if(input[i]==(881+offset0))
{
	vR[i] = 881;
}
else
{
	vR[i] = 880;
}

}

}

}

}
else
{
	if(input[i]>(871+offset0))
{
	if(input[i]>(875+offset0))
{
	if(input[i]>(877+offset0))
{
	if(input[i]==(879+offset0))
{
	vR[i] = 879;
}
else
{
	vR[i] = 878;
}

}
else
{
	if(input[i]==(877+offset0))
{
	vR[i] = 877;
}
else
{
	vR[i] = 876;
}

}

}
else
{
	if(input[i]>(873+offset0))
{
	if(input[i]==(875+offset0))
{
	vR[i] = 875;
}
else
{
	vR[i] = 874;
}

}
else
{
	if(input[i]==(873+offset0))
{
	vR[i] = 873;
}
else
{
	vR[i] = 872;
}

}

}

}
else
{
	if(input[i]>(867+offset0))
{
	if(input[i]>(869+offset0))
{
	if(input[i]==(871+offset0))
{
	vR[i] = 871;
}
else
{
	vR[i] = 870;
}

}
else
{
	if(input[i]==(869+offset0))
{
	vR[i] = 869;
}
else
{
	vR[i] = 868;
}

}

}
else
{
	if(input[i]>(865+offset0))
{
	if(input[i]==(867+offset0))
{
	vR[i] = 867;
}
else
{
	vR[i] = 866;
}

}
else
{
	if(input[i]==(865+offset0))
{
	vR[i] = 865;
}
else
{
	vR[i] = 864;
}

}

}

}

}

}
else
{
	if(input[i]>(847+offset0))
{
	if(input[i]>(855+offset0))
{
	if(input[i]>(859+offset0))
{
	if(input[i]>(861+offset0))
{
	if(input[i]==(863+offset0))
{
	vR[i] = 863;
}
else
{
	vR[i] = 862;
}

}
else
{
	if(input[i]==(861+offset0))
{
	vR[i] = 861;
}
else
{
	vR[i] = 860;
}

}

}
else
{
	if(input[i]>(857+offset0))
{
	if(input[i]==(859+offset0))
{
	vR[i] = 859;
}
else
{
	vR[i] = 858;
}

}
else
{
	if(input[i]==(857+offset0))
{
	vR[i] = 857;
}
else
{
	vR[i] = 856;
}

}

}

}
else
{
	if(input[i]>(851+offset0))
{
	if(input[i]>(853+offset0))
{
	if(input[i]==(855+offset0))
{
	vR[i] = 855;
}
else
{
	vR[i] = 854;
}

}
else
{
	if(input[i]==(853+offset0))
{
	vR[i] = 853;
}
else
{
	vR[i] = 852;
}

}

}
else
{
	if(input[i]>(849+offset0))
{
	if(input[i]==(851+offset0))
{
	vR[i] = 851;
}
else
{
	vR[i] = 850;
}

}
else
{
	if(input[i]==(849+offset0))
{
	vR[i] = 849;
}
else
{
	vR[i] = 848;
}

}

}

}

}
else
{
	if(input[i]>(839+offset0))
{
	if(input[i]>(843+offset0))
{
	if(input[i]>(845+offset0))
{
	if(input[i]==(847+offset0))
{
	vR[i] = 847;
}
else
{
	vR[i] = 846;
}

}
else
{
	if(input[i]==(845+offset0))
{
	vR[i] = 845;
}
else
{
	vR[i] = 844;
}

}

}
else
{
	if(input[i]>(841+offset0))
{
	if(input[i]==(843+offset0))
{
	vR[i] = 843;
}
else
{
	vR[i] = 842;
}

}
else
{
	if(input[i]==(841+offset0))
{
	vR[i] = 841;
}
else
{
	vR[i] = 840;
}

}

}

}
else
{
	if(input[i]>(835+offset0))
{
	if(input[i]>(837+offset0))
{
	if(input[i]==(839+offset0))
{
	vR[i] = 839;
}
else
{
	vR[i] = 838;
}

}
else
{
	if(input[i]==(837+offset0))
{
	vR[i] = 837;
}
else
{
	vR[i] = 836;
}

}

}
else
{
	if(input[i]>(833+offset0))
{
	if(input[i]==(835+offset0))
{
	vR[i] = 835;
}
else
{
	vR[i] = 834;
}

}
else
{
	if(input[i]==(833+offset0))
{
	vR[i] = 833;
}
else
{
	vR[i] = 832;
}

}

}

}

}

}

}
else
{
	if(input[i]>(799+offset0))
{
	if(input[i]>(815+offset0))
{
	if(input[i]>(823+offset0))
{
	if(input[i]>(827+offset0))
{
	if(input[i]>(829+offset0))
{
	if(input[i]==(831+offset0))
{
	vR[i] = 831;
}
else
{
	vR[i] = 830;
}

}
else
{
	if(input[i]==(829+offset0))
{
	vR[i] = 829;
}
else
{
	vR[i] = 828;
}

}

}
else
{
	if(input[i]>(825+offset0))
{
	if(input[i]==(827+offset0))
{
	vR[i] = 827;
}
else
{
	vR[i] = 826;
}

}
else
{
	if(input[i]==(825+offset0))
{
	vR[i] = 825;
}
else
{
	vR[i] = 824;
}

}

}

}
else
{
	if(input[i]>(819+offset0))
{
	if(input[i]>(821+offset0))
{
	if(input[i]==(823+offset0))
{
	vR[i] = 823;
}
else
{
	vR[i] = 822;
}

}
else
{
	if(input[i]==(821+offset0))
{
	vR[i] = 821;
}
else
{
	vR[i] = 820;
}

}

}
else
{
	if(input[i]>(817+offset0))
{
	if(input[i]==(819+offset0))
{
	vR[i] = 819;
}
else
{
	vR[i] = 818;
}

}
else
{
	if(input[i]==(817+offset0))
{
	vR[i] = 817;
}
else
{
	vR[i] = 816;
}

}

}

}

}
else
{
	if(input[i]>(807+offset0))
{
	if(input[i]>(811+offset0))
{
	if(input[i]>(813+offset0))
{
	if(input[i]==(815+offset0))
{
	vR[i] = 815;
}
else
{
	vR[i] = 814;
}

}
else
{
	if(input[i]==(813+offset0))
{
	vR[i] = 813;
}
else
{
	vR[i] = 812;
}

}

}
else
{
	if(input[i]>(809+offset0))
{
	if(input[i]==(811+offset0))
{
	vR[i] = 811;
}
else
{
	vR[i] = 810;
}

}
else
{
	if(input[i]==(809+offset0))
{
	vR[i] = 809;
}
else
{
	vR[i] = 808;
}

}

}

}
else
{
	if(input[i]>(803+offset0))
{
	if(input[i]>(805+offset0))
{
	if(input[i]==(807+offset0))
{
	vR[i] = 807;
}
else
{
	vR[i] = 806;
}

}
else
{
	if(input[i]==(805+offset0))
{
	vR[i] = 805;
}
else
{
	vR[i] = 804;
}

}

}
else
{
	if(input[i]>(801+offset0))
{
	if(input[i]==(803+offset0))
{
	vR[i] = 803;
}
else
{
	vR[i] = 802;
}

}
else
{
	if(input[i]==(801+offset0))
{
	vR[i] = 801;
}
else
{
	vR[i] = 800;
}

}

}

}

}

}
else
{
	if(input[i]>(783+offset0))
{
	if(input[i]>(791+offset0))
{
	if(input[i]>(795+offset0))
{
	if(input[i]>(797+offset0))
{
	if(input[i]==(799+offset0))
{
	vR[i] = 799;
}
else
{
	vR[i] = 798;
}

}
else
{
	if(input[i]==(797+offset0))
{
	vR[i] = 797;
}
else
{
	vR[i] = 796;
}

}

}
else
{
	if(input[i]>(793+offset0))
{
	if(input[i]==(795+offset0))
{
	vR[i] = 795;
}
else
{
	vR[i] = 794;
}

}
else
{
	if(input[i]==(793+offset0))
{
	vR[i] = 793;
}
else
{
	vR[i] = 792;
}

}

}

}
else
{
	if(input[i]>(787+offset0))
{
	if(input[i]>(789+offset0))
{
	if(input[i]==(791+offset0))
{
	vR[i] = 791;
}
else
{
	vR[i] = 790;
}

}
else
{
	if(input[i]==(789+offset0))
{
	vR[i] = 789;
}
else
{
	vR[i] = 788;
}

}

}
else
{
	if(input[i]>(785+offset0))
{
	if(input[i]==(787+offset0))
{
	vR[i] = 787;
}
else
{
	vR[i] = 786;
}

}
else
{
	if(input[i]==(785+offset0))
{
	vR[i] = 785;
}
else
{
	vR[i] = 784;
}

}

}

}

}
else
{
	if(input[i]>(775+offset0))
{
	if(input[i]>(779+offset0))
{
	if(input[i]>(781+offset0))
{
	if(input[i]==(783+offset0))
{
	vR[i] = 783;
}
else
{
	vR[i] = 782;
}

}
else
{
	if(input[i]==(781+offset0))
{
	vR[i] = 781;
}
else
{
	vR[i] = 780;
}

}

}
else
{
	if(input[i]>(777+offset0))
{
	if(input[i]==(779+offset0))
{
	vR[i] = 779;
}
else
{
	vR[i] = 778;
}

}
else
{
	if(input[i]==(777+offset0))
{
	vR[i] = 777;
}
else
{
	vR[i] = 776;
}

}

}

}
else
{
	if(input[i]>(771+offset0))
{
	if(input[i]>(773+offset0))
{
	if(input[i]==(775+offset0))
{
	vR[i] = 775;
}
else
{
	vR[i] = 774;
}

}
else
{
	if(input[i]==(773+offset0))
{
	vR[i] = 773;
}
else
{
	vR[i] = 772;
}

}

}
else
{
	if(input[i]>(769+offset0))
{
	if(input[i]==(771+offset0))
{
	vR[i] = 771;
}
else
{
	vR[i] = 770;
}

}
else
{
	if(input[i]==(769+offset0))
{
	vR[i] = 769;
}
else
{
	vR[i] = 768;
}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(639+offset0))
{
	if(input[i]>(703+offset0))
{
	if(input[i]>(735+offset0))
{
	if(input[i]>(751+offset0))
{
	if(input[i]>(759+offset0))
{
	if(input[i]>(763+offset0))
{
	if(input[i]>(765+offset0))
{
	if(input[i]==(767+offset0))
{
	vR[i] = 767;
}
else
{
	vR[i] = 766;
}

}
else
{
	if(input[i]==(765+offset0))
{
	vR[i] = 765;
}
else
{
	vR[i] = 764;
}

}

}
else
{
	if(input[i]>(761+offset0))
{
	if(input[i]==(763+offset0))
{
	vR[i] = 763;
}
else
{
	vR[i] = 762;
}

}
else
{
	if(input[i]==(761+offset0))
{
	vR[i] = 761;
}
else
{
	vR[i] = 760;
}

}

}

}
else
{
	if(input[i]>(755+offset0))
{
	if(input[i]>(757+offset0))
{
	if(input[i]==(759+offset0))
{
	vR[i] = 759;
}
else
{
	vR[i] = 758;
}

}
else
{
	if(input[i]==(757+offset0))
{
	vR[i] = 757;
}
else
{
	vR[i] = 756;
}

}

}
else
{
	if(input[i]>(753+offset0))
{
	if(input[i]==(755+offset0))
{
	vR[i] = 755;
}
else
{
	vR[i] = 754;
}

}
else
{
	if(input[i]==(753+offset0))
{
	vR[i] = 753;
}
else
{
	vR[i] = 752;
}

}

}

}

}
else
{
	if(input[i]>(743+offset0))
{
	if(input[i]>(747+offset0))
{
	if(input[i]>(749+offset0))
{
	if(input[i]==(751+offset0))
{
	vR[i] = 751;
}
else
{
	vR[i] = 750;
}

}
else
{
	if(input[i]==(749+offset0))
{
	vR[i] = 749;
}
else
{
	vR[i] = 748;
}

}

}
else
{
	if(input[i]>(745+offset0))
{
	if(input[i]==(747+offset0))
{
	vR[i] = 747;
}
else
{
	vR[i] = 746;
}

}
else
{
	if(input[i]==(745+offset0))
{
	vR[i] = 745;
}
else
{
	vR[i] = 744;
}

}

}

}
else
{
	if(input[i]>(739+offset0))
{
	if(input[i]>(741+offset0))
{
	if(input[i]==(743+offset0))
{
	vR[i] = 743;
}
else
{
	vR[i] = 742;
}

}
else
{
	if(input[i]==(741+offset0))
{
	vR[i] = 741;
}
else
{
	vR[i] = 740;
}

}

}
else
{
	if(input[i]>(737+offset0))
{
	if(input[i]==(739+offset0))
{
	vR[i] = 739;
}
else
{
	vR[i] = 738;
}

}
else
{
	if(input[i]==(737+offset0))
{
	vR[i] = 737;
}
else
{
	vR[i] = 736;
}

}

}

}

}

}
else
{
	if(input[i]>(719+offset0))
{
	if(input[i]>(727+offset0))
{
	if(input[i]>(731+offset0))
{
	if(input[i]>(733+offset0))
{
	if(input[i]==(735+offset0))
{
	vR[i] = 735;
}
else
{
	vR[i] = 734;
}

}
else
{
	if(input[i]==(733+offset0))
{
	vR[i] = 733;
}
else
{
	vR[i] = 732;
}

}

}
else
{
	if(input[i]>(729+offset0))
{
	if(input[i]==(731+offset0))
{
	vR[i] = 731;
}
else
{
	vR[i] = 730;
}

}
else
{
	if(input[i]==(729+offset0))
{
	vR[i] = 729;
}
else
{
	vR[i] = 728;
}

}

}

}
else
{
	if(input[i]>(723+offset0))
{
	if(input[i]>(725+offset0))
{
	if(input[i]==(727+offset0))
{
	vR[i] = 727;
}
else
{
	vR[i] = 726;
}

}
else
{
	if(input[i]==(725+offset0))
{
	vR[i] = 725;
}
else
{
	vR[i] = 724;
}

}

}
else
{
	if(input[i]>(721+offset0))
{
	if(input[i]==(723+offset0))
{
	vR[i] = 723;
}
else
{
	vR[i] = 722;
}

}
else
{
	if(input[i]==(721+offset0))
{
	vR[i] = 721;
}
else
{
	vR[i] = 720;
}

}

}

}

}
else
{
	if(input[i]>(711+offset0))
{
	if(input[i]>(715+offset0))
{
	if(input[i]>(717+offset0))
{
	if(input[i]==(719+offset0))
{
	vR[i] = 719;
}
else
{
	vR[i] = 718;
}

}
else
{
	if(input[i]==(717+offset0))
{
	vR[i] = 717;
}
else
{
	vR[i] = 716;
}

}

}
else
{
	if(input[i]>(713+offset0))
{
	if(input[i]==(715+offset0))
{
	vR[i] = 715;
}
else
{
	vR[i] = 714;
}

}
else
{
	if(input[i]==(713+offset0))
{
	vR[i] = 713;
}
else
{
	vR[i] = 712;
}

}

}

}
else
{
	if(input[i]>(707+offset0))
{
	if(input[i]>(709+offset0))
{
	if(input[i]==(711+offset0))
{
	vR[i] = 711;
}
else
{
	vR[i] = 710;
}

}
else
{
	if(input[i]==(709+offset0))
{
	vR[i] = 709;
}
else
{
	vR[i] = 708;
}

}

}
else
{
	if(input[i]>(705+offset0))
{
	if(input[i]==(707+offset0))
{
	vR[i] = 707;
}
else
{
	vR[i] = 706;
}

}
else
{
	if(input[i]==(705+offset0))
{
	vR[i] = 705;
}
else
{
	vR[i] = 704;
}

}

}

}

}

}

}
else
{
	if(input[i]>(671+offset0))
{
	if(input[i]>(687+offset0))
{
	if(input[i]>(695+offset0))
{
	if(input[i]>(699+offset0))
{
	if(input[i]>(701+offset0))
{
	if(input[i]==(703+offset0))
{
	vR[i] = 703;
}
else
{
	vR[i] = 702;
}

}
else
{
	if(input[i]==(701+offset0))
{
	vR[i] = 701;
}
else
{
	vR[i] = 700;
}

}

}
else
{
	if(input[i]>(697+offset0))
{
	if(input[i]==(699+offset0))
{
	vR[i] = 699;
}
else
{
	vR[i] = 698;
}

}
else
{
	if(input[i]==(697+offset0))
{
	vR[i] = 697;
}
else
{
	vR[i] = 696;
}

}

}

}
else
{
	if(input[i]>(691+offset0))
{
	if(input[i]>(693+offset0))
{
	if(input[i]==(695+offset0))
{
	vR[i] = 695;
}
else
{
	vR[i] = 694;
}

}
else
{
	if(input[i]==(693+offset0))
{
	vR[i] = 693;
}
else
{
	vR[i] = 692;
}

}

}
else
{
	if(input[i]>(689+offset0))
{
	if(input[i]==(691+offset0))
{
	vR[i] = 691;
}
else
{
	vR[i] = 690;
}

}
else
{
	if(input[i]==(689+offset0))
{
	vR[i] = 689;
}
else
{
	vR[i] = 688;
}

}

}

}

}
else
{
	if(input[i]>(679+offset0))
{
	if(input[i]>(683+offset0))
{
	if(input[i]>(685+offset0))
{
	if(input[i]==(687+offset0))
{
	vR[i] = 687;
}
else
{
	vR[i] = 686;
}

}
else
{
	if(input[i]==(685+offset0))
{
	vR[i] = 685;
}
else
{
	vR[i] = 684;
}

}

}
else
{
	if(input[i]>(681+offset0))
{
	if(input[i]==(683+offset0))
{
	vR[i] = 683;
}
else
{
	vR[i] = 682;
}

}
else
{
	if(input[i]==(681+offset0))
{
	vR[i] = 681;
}
else
{
	vR[i] = 680;
}

}

}

}
else
{
	if(input[i]>(675+offset0))
{
	if(input[i]>(677+offset0))
{
	if(input[i]==(679+offset0))
{
	vR[i] = 679;
}
else
{
	vR[i] = 678;
}

}
else
{
	if(input[i]==(677+offset0))
{
	vR[i] = 677;
}
else
{
	vR[i] = 676;
}

}

}
else
{
	if(input[i]>(673+offset0))
{
	if(input[i]==(675+offset0))
{
	vR[i] = 675;
}
else
{
	vR[i] = 674;
}

}
else
{
	if(input[i]==(673+offset0))
{
	vR[i] = 673;
}
else
{
	vR[i] = 672;
}

}

}

}

}

}
else
{
	if(input[i]>(655+offset0))
{
	if(input[i]>(663+offset0))
{
	if(input[i]>(667+offset0))
{
	if(input[i]>(669+offset0))
{
	if(input[i]==(671+offset0))
{
	vR[i] = 671;
}
else
{
	vR[i] = 670;
}

}
else
{
	if(input[i]==(669+offset0))
{
	vR[i] = 669;
}
else
{
	vR[i] = 668;
}

}

}
else
{
	if(input[i]>(665+offset0))
{
	if(input[i]==(667+offset0))
{
	vR[i] = 667;
}
else
{
	vR[i] = 666;
}

}
else
{
	if(input[i]==(665+offset0))
{
	vR[i] = 665;
}
else
{
	vR[i] = 664;
}

}

}

}
else
{
	if(input[i]>(659+offset0))
{
	if(input[i]>(661+offset0))
{
	if(input[i]==(663+offset0))
{
	vR[i] = 663;
}
else
{
	vR[i] = 662;
}

}
else
{
	if(input[i]==(661+offset0))
{
	vR[i] = 661;
}
else
{
	vR[i] = 660;
}

}

}
else
{
	if(input[i]>(657+offset0))
{
	if(input[i]==(659+offset0))
{
	vR[i] = 659;
}
else
{
	vR[i] = 658;
}

}
else
{
	if(input[i]==(657+offset0))
{
	vR[i] = 657;
}
else
{
	vR[i] = 656;
}

}

}

}

}
else
{
	if(input[i]>(647+offset0))
{
	if(input[i]>(651+offset0))
{
	if(input[i]>(653+offset0))
{
	if(input[i]==(655+offset0))
{
	vR[i] = 655;
}
else
{
	vR[i] = 654;
}

}
else
{
	if(input[i]==(653+offset0))
{
	vR[i] = 653;
}
else
{
	vR[i] = 652;
}

}

}
else
{
	if(input[i]>(649+offset0))
{
	if(input[i]==(651+offset0))
{
	vR[i] = 651;
}
else
{
	vR[i] = 650;
}

}
else
{
	if(input[i]==(649+offset0))
{
	vR[i] = 649;
}
else
{
	vR[i] = 648;
}

}

}

}
else
{
	if(input[i]>(643+offset0))
{
	if(input[i]>(645+offset0))
{
	if(input[i]==(647+offset0))
{
	vR[i] = 647;
}
else
{
	vR[i] = 646;
}

}
else
{
	if(input[i]==(645+offset0))
{
	vR[i] = 645;
}
else
{
	vR[i] = 644;
}

}

}
else
{
	if(input[i]>(641+offset0))
{
	if(input[i]==(643+offset0))
{
	vR[i] = 643;
}
else
{
	vR[i] = 642;
}

}
else
{
	if(input[i]==(641+offset0))
{
	vR[i] = 641;
}
else
{
	vR[i] = 640;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(575+offset0))
{
	if(input[i]>(607+offset0))
{
	if(input[i]>(623+offset0))
{
	if(input[i]>(631+offset0))
{
	if(input[i]>(635+offset0))
{
	if(input[i]>(637+offset0))
{
	if(input[i]==(639+offset0))
{
	vR[i] = 639;
}
else
{
	vR[i] = 638;
}

}
else
{
	if(input[i]==(637+offset0))
{
	vR[i] = 637;
}
else
{
	vR[i] = 636;
}

}

}
else
{
	if(input[i]>(633+offset0))
{
	if(input[i]==(635+offset0))
{
	vR[i] = 635;
}
else
{
	vR[i] = 634;
}

}
else
{
	if(input[i]==(633+offset0))
{
	vR[i] = 633;
}
else
{
	vR[i] = 632;
}

}

}

}
else
{
	if(input[i]>(627+offset0))
{
	if(input[i]>(629+offset0))
{
	if(input[i]==(631+offset0))
{
	vR[i] = 631;
}
else
{
	vR[i] = 630;
}

}
else
{
	if(input[i]==(629+offset0))
{
	vR[i] = 629;
}
else
{
	vR[i] = 628;
}

}

}
else
{
	if(input[i]>(625+offset0))
{
	if(input[i]==(627+offset0))
{
	vR[i] = 627;
}
else
{
	vR[i] = 626;
}

}
else
{
	if(input[i]==(625+offset0))
{
	vR[i] = 625;
}
else
{
	vR[i] = 624;
}

}

}

}

}
else
{
	if(input[i]>(615+offset0))
{
	if(input[i]>(619+offset0))
{
	if(input[i]>(621+offset0))
{
	if(input[i]==(623+offset0))
{
	vR[i] = 623;
}
else
{
	vR[i] = 622;
}

}
else
{
	if(input[i]==(621+offset0))
{
	vR[i] = 621;
}
else
{
	vR[i] = 620;
}

}

}
else
{
	if(input[i]>(617+offset0))
{
	if(input[i]==(619+offset0))
{
	vR[i] = 619;
}
else
{
	vR[i] = 618;
}

}
else
{
	if(input[i]==(617+offset0))
{
	vR[i] = 617;
}
else
{
	vR[i] = 616;
}

}

}

}
else
{
	if(input[i]>(611+offset0))
{
	if(input[i]>(613+offset0))
{
	if(input[i]==(615+offset0))
{
	vR[i] = 615;
}
else
{
	vR[i] = 614;
}

}
else
{
	if(input[i]==(613+offset0))
{
	vR[i] = 613;
}
else
{
	vR[i] = 612;
}

}

}
else
{
	if(input[i]>(609+offset0))
{
	if(input[i]==(611+offset0))
{
	vR[i] = 611;
}
else
{
	vR[i] = 610;
}

}
else
{
	if(input[i]==(609+offset0))
{
	vR[i] = 609;
}
else
{
	vR[i] = 608;
}

}

}

}

}

}
else
{
	if(input[i]>(591+offset0))
{
	if(input[i]>(599+offset0))
{
	if(input[i]>(603+offset0))
{
	if(input[i]>(605+offset0))
{
	if(input[i]==(607+offset0))
{
	vR[i] = 607;
}
else
{
	vR[i] = 606;
}

}
else
{
	if(input[i]==(605+offset0))
{
	vR[i] = 605;
}
else
{
	vR[i] = 604;
}

}

}
else
{
	if(input[i]>(601+offset0))
{
	if(input[i]==(603+offset0))
{
	vR[i] = 603;
}
else
{
	vR[i] = 602;
}

}
else
{
	if(input[i]==(601+offset0))
{
	vR[i] = 601;
}
else
{
	vR[i] = 600;
}

}

}

}
else
{
	if(input[i]>(595+offset0))
{
	if(input[i]>(597+offset0))
{
	if(input[i]==(599+offset0))
{
	vR[i] = 599;
}
else
{
	vR[i] = 598;
}

}
else
{
	if(input[i]==(597+offset0))
{
	vR[i] = 597;
}
else
{
	vR[i] = 596;
}

}

}
else
{
	if(input[i]>(593+offset0))
{
	if(input[i]==(595+offset0))
{
	vR[i] = 595;
}
else
{
	vR[i] = 594;
}

}
else
{
	if(input[i]==(593+offset0))
{
	vR[i] = 593;
}
else
{
	vR[i] = 592;
}

}

}

}

}
else
{
	if(input[i]>(583+offset0))
{
	if(input[i]>(587+offset0))
{
	if(input[i]>(589+offset0))
{
	if(input[i]==(591+offset0))
{
	vR[i] = 591;
}
else
{
	vR[i] = 590;
}

}
else
{
	if(input[i]==(589+offset0))
{
	vR[i] = 589;
}
else
{
	vR[i] = 588;
}

}

}
else
{
	if(input[i]>(585+offset0))
{
	if(input[i]==(587+offset0))
{
	vR[i] = 587;
}
else
{
	vR[i] = 586;
}

}
else
{
	if(input[i]==(585+offset0))
{
	vR[i] = 585;
}
else
{
	vR[i] = 584;
}

}

}

}
else
{
	if(input[i]>(579+offset0))
{
	if(input[i]>(581+offset0))
{
	if(input[i]==(583+offset0))
{
	vR[i] = 583;
}
else
{
	vR[i] = 582;
}

}
else
{
	if(input[i]==(581+offset0))
{
	vR[i] = 581;
}
else
{
	vR[i] = 580;
}

}

}
else
{
	if(input[i]>(577+offset0))
{
	if(input[i]==(579+offset0))
{
	vR[i] = 579;
}
else
{
	vR[i] = 578;
}

}
else
{
	if(input[i]==(577+offset0))
{
	vR[i] = 577;
}
else
{
	vR[i] = 576;
}

}

}

}

}

}

}
else
{
	if(input[i]>(543+offset0))
{
	if(input[i]>(559+offset0))
{
	if(input[i]>(567+offset0))
{
	if(input[i]>(571+offset0))
{
	if(input[i]>(573+offset0))
{
	if(input[i]==(575+offset0))
{
	vR[i] = 575;
}
else
{
	vR[i] = 574;
}

}
else
{
	if(input[i]==(573+offset0))
{
	vR[i] = 573;
}
else
{
	vR[i] = 572;
}

}

}
else
{
	if(input[i]>(569+offset0))
{
	if(input[i]==(571+offset0))
{
	vR[i] = 571;
}
else
{
	vR[i] = 570;
}

}
else
{
	if(input[i]==(569+offset0))
{
	vR[i] = 569;
}
else
{
	vR[i] = 568;
}

}

}

}
else
{
	if(input[i]>(563+offset0))
{
	if(input[i]>(565+offset0))
{
	if(input[i]==(567+offset0))
{
	vR[i] = 567;
}
else
{
	vR[i] = 566;
}

}
else
{
	if(input[i]==(565+offset0))
{
	vR[i] = 565;
}
else
{
	vR[i] = 564;
}

}

}
else
{
	if(input[i]>(561+offset0))
{
	if(input[i]==(563+offset0))
{
	vR[i] = 563;
}
else
{
	vR[i] = 562;
}

}
else
{
	if(input[i]==(561+offset0))
{
	vR[i] = 561;
}
else
{
	vR[i] = 560;
}

}

}

}

}
else
{
	if(input[i]>(551+offset0))
{
	if(input[i]>(555+offset0))
{
	if(input[i]>(557+offset0))
{
	if(input[i]==(559+offset0))
{
	vR[i] = 559;
}
else
{
	vR[i] = 558;
}

}
else
{
	if(input[i]==(557+offset0))
{
	vR[i] = 557;
}
else
{
	vR[i] = 556;
}

}

}
else
{
	if(input[i]>(553+offset0))
{
	if(input[i]==(555+offset0))
{
	vR[i] = 555;
}
else
{
	vR[i] = 554;
}

}
else
{
	if(input[i]==(553+offset0))
{
	vR[i] = 553;
}
else
{
	vR[i] = 552;
}

}

}

}
else
{
	if(input[i]>(547+offset0))
{
	if(input[i]>(549+offset0))
{
	if(input[i]==(551+offset0))
{
	vR[i] = 551;
}
else
{
	vR[i] = 550;
}

}
else
{
	if(input[i]==(549+offset0))
{
	vR[i] = 549;
}
else
{
	vR[i] = 548;
}

}

}
else
{
	if(input[i]>(545+offset0))
{
	if(input[i]==(547+offset0))
{
	vR[i] = 547;
}
else
{
	vR[i] = 546;
}

}
else
{
	if(input[i]==(545+offset0))
{
	vR[i] = 545;
}
else
{
	vR[i] = 544;
}

}

}

}

}

}
else
{
	if(input[i]>(527+offset0))
{
	if(input[i]>(535+offset0))
{
	if(input[i]>(539+offset0))
{
	if(input[i]>(541+offset0))
{
	if(input[i]==(543+offset0))
{
	vR[i] = 543;
}
else
{
	vR[i] = 542;
}

}
else
{
	if(input[i]==(541+offset0))
{
	vR[i] = 541;
}
else
{
	vR[i] = 540;
}

}

}
else
{
	if(input[i]>(537+offset0))
{
	if(input[i]==(539+offset0))
{
	vR[i] = 539;
}
else
{
	vR[i] = 538;
}

}
else
{
	if(input[i]==(537+offset0))
{
	vR[i] = 537;
}
else
{
	vR[i] = 536;
}

}

}

}
else
{
	if(input[i]>(531+offset0))
{
	if(input[i]>(533+offset0))
{
	if(input[i]==(535+offset0))
{
	vR[i] = 535;
}
else
{
	vR[i] = 534;
}

}
else
{
	if(input[i]==(533+offset0))
{
	vR[i] = 533;
}
else
{
	vR[i] = 532;
}

}

}
else
{
	if(input[i]>(529+offset0))
{
	if(input[i]==(531+offset0))
{
	vR[i] = 531;
}
else
{
	vR[i] = 530;
}

}
else
{
	if(input[i]==(529+offset0))
{
	vR[i] = 529;
}
else
{
	vR[i] = 528;
}

}

}

}

}
else
{
	if(input[i]>(519+offset0))
{
	if(input[i]>(523+offset0))
{
	if(input[i]>(525+offset0))
{
	if(input[i]==(527+offset0))
{
	vR[i] = 527;
}
else
{
	vR[i] = 526;
}

}
else
{
	if(input[i]==(525+offset0))
{
	vR[i] = 525;
}
else
{
	vR[i] = 524;
}

}

}
else
{
	if(input[i]>(521+offset0))
{
	if(input[i]==(523+offset0))
{
	vR[i] = 523;
}
else
{
	vR[i] = 522;
}

}
else
{
	if(input[i]==(521+offset0))
{
	vR[i] = 521;
}
else
{
	vR[i] = 520;
}

}

}

}
else
{
	if(input[i]>(515+offset0))
{
	if(input[i]>(517+offset0))
{
	if(input[i]==(519+offset0))
{
	vR[i] = 519;
}
else
{
	vR[i] = 518;
}

}
else
{
	if(input[i]==(517+offset0))
{
	vR[i] = 517;
}
else
{
	vR[i] = 516;
}

}

}
else
{
	if(input[i]>(513+offset0))
{
	if(input[i]==(515+offset0))
{
	vR[i] = 515;
}
else
{
	vR[i] = 514;
}

}
else
{
	if(input[i]==(513+offset0))
{
	vR[i] = 513;
}
else
{
	vR[i] = 512;
}

}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(255+offset0))
{
	if(input[i]>(383+offset0))
{
	if(input[i]>(447+offset0))
{
	if(input[i]>(479+offset0))
{
	if(input[i]>(495+offset0))
{
	if(input[i]>(503+offset0))
{
	if(input[i]>(507+offset0))
{
	if(input[i]>(509+offset0))
{
	if(input[i]==(511+offset0))
{
	vR[i] = 511;
}
else
{
	vR[i] = 510;
}

}
else
{
	if(input[i]==(509+offset0))
{
	vR[i] = 509;
}
else
{
	vR[i] = 508;
}

}

}
else
{
	if(input[i]>(505+offset0))
{
	if(input[i]==(507+offset0))
{
	vR[i] = 507;
}
else
{
	vR[i] = 506;
}

}
else
{
	if(input[i]==(505+offset0))
{
	vR[i] = 505;
}
else
{
	vR[i] = 504;
}

}

}

}
else
{
	if(input[i]>(499+offset0))
{
	if(input[i]>(501+offset0))
{
	if(input[i]==(503+offset0))
{
	vR[i] = 503;
}
else
{
	vR[i] = 502;
}

}
else
{
	if(input[i]==(501+offset0))
{
	vR[i] = 501;
}
else
{
	vR[i] = 500;
}

}

}
else
{
	if(input[i]>(497+offset0))
{
	if(input[i]==(499+offset0))
{
	vR[i] = 499;
}
else
{
	vR[i] = 498;
}

}
else
{
	if(input[i]==(497+offset0))
{
	vR[i] = 497;
}
else
{
	vR[i] = 496;
}

}

}

}

}
else
{
	if(input[i]>(487+offset0))
{
	if(input[i]>(491+offset0))
{
	if(input[i]>(493+offset0))
{
	if(input[i]==(495+offset0))
{
	vR[i] = 495;
}
else
{
	vR[i] = 494;
}

}
else
{
	if(input[i]==(493+offset0))
{
	vR[i] = 493;
}
else
{
	vR[i] = 492;
}

}

}
else
{
	if(input[i]>(489+offset0))
{
	if(input[i]==(491+offset0))
{
	vR[i] = 491;
}
else
{
	vR[i] = 490;
}

}
else
{
	if(input[i]==(489+offset0))
{
	vR[i] = 489;
}
else
{
	vR[i] = 488;
}

}

}

}
else
{
	if(input[i]>(483+offset0))
{
	if(input[i]>(485+offset0))
{
	if(input[i]==(487+offset0))
{
	vR[i] = 487;
}
else
{
	vR[i] = 486;
}

}
else
{
	if(input[i]==(485+offset0))
{
	vR[i] = 485;
}
else
{
	vR[i] = 484;
}

}

}
else
{
	if(input[i]>(481+offset0))
{
	if(input[i]==(483+offset0))
{
	vR[i] = 483;
}
else
{
	vR[i] = 482;
}

}
else
{
	if(input[i]==(481+offset0))
{
	vR[i] = 481;
}
else
{
	vR[i] = 480;
}

}

}

}

}

}
else
{
	if(input[i]>(463+offset0))
{
	if(input[i]>(471+offset0))
{
	if(input[i]>(475+offset0))
{
	if(input[i]>(477+offset0))
{
	if(input[i]==(479+offset0))
{
	vR[i] = 479;
}
else
{
	vR[i] = 478;
}

}
else
{
	if(input[i]==(477+offset0))
{
	vR[i] = 477;
}
else
{
	vR[i] = 476;
}

}

}
else
{
	if(input[i]>(473+offset0))
{
	if(input[i]==(475+offset0))
{
	vR[i] = 475;
}
else
{
	vR[i] = 474;
}

}
else
{
	if(input[i]==(473+offset0))
{
	vR[i] = 473;
}
else
{
	vR[i] = 472;
}

}

}

}
else
{
	if(input[i]>(467+offset0))
{
	if(input[i]>(469+offset0))
{
	if(input[i]==(471+offset0))
{
	vR[i] = 471;
}
else
{
	vR[i] = 470;
}

}
else
{
	if(input[i]==(469+offset0))
{
	vR[i] = 469;
}
else
{
	vR[i] = 468;
}

}

}
else
{
	if(input[i]>(465+offset0))
{
	if(input[i]==(467+offset0))
{
	vR[i] = 467;
}
else
{
	vR[i] = 466;
}

}
else
{
	if(input[i]==(465+offset0))
{
	vR[i] = 465;
}
else
{
	vR[i] = 464;
}

}

}

}

}
else
{
	if(input[i]>(455+offset0))
{
	if(input[i]>(459+offset0))
{
	if(input[i]>(461+offset0))
{
	if(input[i]==(463+offset0))
{
	vR[i] = 463;
}
else
{
	vR[i] = 462;
}

}
else
{
	if(input[i]==(461+offset0))
{
	vR[i] = 461;
}
else
{
	vR[i] = 460;
}

}

}
else
{
	if(input[i]>(457+offset0))
{
	if(input[i]==(459+offset0))
{
	vR[i] = 459;
}
else
{
	vR[i] = 458;
}

}
else
{
	if(input[i]==(457+offset0))
{
	vR[i] = 457;
}
else
{
	vR[i] = 456;
}

}

}

}
else
{
	if(input[i]>(451+offset0))
{
	if(input[i]>(453+offset0))
{
	if(input[i]==(455+offset0))
{
	vR[i] = 455;
}
else
{
	vR[i] = 454;
}

}
else
{
	if(input[i]==(453+offset0))
{
	vR[i] = 453;
}
else
{
	vR[i] = 452;
}

}

}
else
{
	if(input[i]>(449+offset0))
{
	if(input[i]==(451+offset0))
{
	vR[i] = 451;
}
else
{
	vR[i] = 450;
}

}
else
{
	if(input[i]==(449+offset0))
{
	vR[i] = 449;
}
else
{
	vR[i] = 448;
}

}

}

}

}

}

}
else
{
	if(input[i]>(415+offset0))
{
	if(input[i]>(431+offset0))
{
	if(input[i]>(439+offset0))
{
	if(input[i]>(443+offset0))
{
	if(input[i]>(445+offset0))
{
	if(input[i]==(447+offset0))
{
	vR[i] = 447;
}
else
{
	vR[i] = 446;
}

}
else
{
	if(input[i]==(445+offset0))
{
	vR[i] = 445;
}
else
{
	vR[i] = 444;
}

}

}
else
{
	if(input[i]>(441+offset0))
{
	if(input[i]==(443+offset0))
{
	vR[i] = 443;
}
else
{
	vR[i] = 442;
}

}
else
{
	if(input[i]==(441+offset0))
{
	vR[i] = 441;
}
else
{
	vR[i] = 440;
}

}

}

}
else
{
	if(input[i]>(435+offset0))
{
	if(input[i]>(437+offset0))
{
	if(input[i]==(439+offset0))
{
	vR[i] = 439;
}
else
{
	vR[i] = 438;
}

}
else
{
	if(input[i]==(437+offset0))
{
	vR[i] = 437;
}
else
{
	vR[i] = 436;
}

}

}
else
{
	if(input[i]>(433+offset0))
{
	if(input[i]==(435+offset0))
{
	vR[i] = 435;
}
else
{
	vR[i] = 434;
}

}
else
{
	if(input[i]==(433+offset0))
{
	vR[i] = 433;
}
else
{
	vR[i] = 432;
}

}

}

}

}
else
{
	if(input[i]>(423+offset0))
{
	if(input[i]>(427+offset0))
{
	if(input[i]>(429+offset0))
{
	if(input[i]==(431+offset0))
{
	vR[i] = 431;
}
else
{
	vR[i] = 430;
}

}
else
{
	if(input[i]==(429+offset0))
{
	vR[i] = 429;
}
else
{
	vR[i] = 428;
}

}

}
else
{
	if(input[i]>(425+offset0))
{
	if(input[i]==(427+offset0))
{
	vR[i] = 427;
}
else
{
	vR[i] = 426;
}

}
else
{
	if(input[i]==(425+offset0))
{
	vR[i] = 425;
}
else
{
	vR[i] = 424;
}

}

}

}
else
{
	if(input[i]>(419+offset0))
{
	if(input[i]>(421+offset0))
{
	if(input[i]==(423+offset0))
{
	vR[i] = 423;
}
else
{
	vR[i] = 422;
}

}
else
{
	if(input[i]==(421+offset0))
{
	vR[i] = 421;
}
else
{
	vR[i] = 420;
}

}

}
else
{
	if(input[i]>(417+offset0))
{
	if(input[i]==(419+offset0))
{
	vR[i] = 419;
}
else
{
	vR[i] = 418;
}

}
else
{
	if(input[i]==(417+offset0))
{
	vR[i] = 417;
}
else
{
	vR[i] = 416;
}

}

}

}

}

}
else
{
	if(input[i]>(399+offset0))
{
	if(input[i]>(407+offset0))
{
	if(input[i]>(411+offset0))
{
	if(input[i]>(413+offset0))
{
	if(input[i]==(415+offset0))
{
	vR[i] = 415;
}
else
{
	vR[i] = 414;
}

}
else
{
	if(input[i]==(413+offset0))
{
	vR[i] = 413;
}
else
{
	vR[i] = 412;
}

}

}
else
{
	if(input[i]>(409+offset0))
{
	if(input[i]==(411+offset0))
{
	vR[i] = 411;
}
else
{
	vR[i] = 410;
}

}
else
{
	if(input[i]==(409+offset0))
{
	vR[i] = 409;
}
else
{
	vR[i] = 408;
}

}

}

}
else
{
	if(input[i]>(403+offset0))
{
	if(input[i]>(405+offset0))
{
	if(input[i]==(407+offset0))
{
	vR[i] = 407;
}
else
{
	vR[i] = 406;
}

}
else
{
	if(input[i]==(405+offset0))
{
	vR[i] = 405;
}
else
{
	vR[i] = 404;
}

}

}
else
{
	if(input[i]>(401+offset0))
{
	if(input[i]==(403+offset0))
{
	vR[i] = 403;
}
else
{
	vR[i] = 402;
}

}
else
{
	if(input[i]==(401+offset0))
{
	vR[i] = 401;
}
else
{
	vR[i] = 400;
}

}

}

}

}
else
{
	if(input[i]>(391+offset0))
{
	if(input[i]>(395+offset0))
{
	if(input[i]>(397+offset0))
{
	if(input[i]==(399+offset0))
{
	vR[i] = 399;
}
else
{
	vR[i] = 398;
}

}
else
{
	if(input[i]==(397+offset0))
{
	vR[i] = 397;
}
else
{
	vR[i] = 396;
}

}

}
else
{
	if(input[i]>(393+offset0))
{
	if(input[i]==(395+offset0))
{
	vR[i] = 395;
}
else
{
	vR[i] = 394;
}

}
else
{
	if(input[i]==(393+offset0))
{
	vR[i] = 393;
}
else
{
	vR[i] = 392;
}

}

}

}
else
{
	if(input[i]>(387+offset0))
{
	if(input[i]>(389+offset0))
{
	if(input[i]==(391+offset0))
{
	vR[i] = 391;
}
else
{
	vR[i] = 390;
}

}
else
{
	if(input[i]==(389+offset0))
{
	vR[i] = 389;
}
else
{
	vR[i] = 388;
}

}

}
else
{
	if(input[i]>(385+offset0))
{
	if(input[i]==(387+offset0))
{
	vR[i] = 387;
}
else
{
	vR[i] = 386;
}

}
else
{
	if(input[i]==(385+offset0))
{
	vR[i] = 385;
}
else
{
	vR[i] = 384;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(319+offset0))
{
	if(input[i]>(351+offset0))
{
	if(input[i]>(367+offset0))
{
	if(input[i]>(375+offset0))
{
	if(input[i]>(379+offset0))
{
	if(input[i]>(381+offset0))
{
	if(input[i]==(383+offset0))
{
	vR[i] = 383;
}
else
{
	vR[i] = 382;
}

}
else
{
	if(input[i]==(381+offset0))
{
	vR[i] = 381;
}
else
{
	vR[i] = 380;
}

}

}
else
{
	if(input[i]>(377+offset0))
{
	if(input[i]==(379+offset0))
{
	vR[i] = 379;
}
else
{
	vR[i] = 378;
}

}
else
{
	if(input[i]==(377+offset0))
{
	vR[i] = 377;
}
else
{
	vR[i] = 376;
}

}

}

}
else
{
	if(input[i]>(371+offset0))
{
	if(input[i]>(373+offset0))
{
	if(input[i]==(375+offset0))
{
	vR[i] = 375;
}
else
{
	vR[i] = 374;
}

}
else
{
	if(input[i]==(373+offset0))
{
	vR[i] = 373;
}
else
{
	vR[i] = 372;
}

}

}
else
{
	if(input[i]>(369+offset0))
{
	if(input[i]==(371+offset0))
{
	vR[i] = 371;
}
else
{
	vR[i] = 370;
}

}
else
{
	if(input[i]==(369+offset0))
{
	vR[i] = 369;
}
else
{
	vR[i] = 368;
}

}

}

}

}
else
{
	if(input[i]>(359+offset0))
{
	if(input[i]>(363+offset0))
{
	if(input[i]>(365+offset0))
{
	if(input[i]==(367+offset0))
{
	vR[i] = 367;
}
else
{
	vR[i] = 366;
}

}
else
{
	if(input[i]==(365+offset0))
{
	vR[i] = 365;
}
else
{
	vR[i] = 364;
}

}

}
else
{
	if(input[i]>(361+offset0))
{
	if(input[i]==(363+offset0))
{
	vR[i] = 363;
}
else
{
	vR[i] = 362;
}

}
else
{
	if(input[i]==(361+offset0))
{
	vR[i] = 361;
}
else
{
	vR[i] = 360;
}

}

}

}
else
{
	if(input[i]>(355+offset0))
{
	if(input[i]>(357+offset0))
{
	if(input[i]==(359+offset0))
{
	vR[i] = 359;
}
else
{
	vR[i] = 358;
}

}
else
{
	if(input[i]==(357+offset0))
{
	vR[i] = 357;
}
else
{
	vR[i] = 356;
}

}

}
else
{
	if(input[i]>(353+offset0))
{
	if(input[i]==(355+offset0))
{
	vR[i] = 355;
}
else
{
	vR[i] = 354;
}

}
else
{
	if(input[i]==(353+offset0))
{
	vR[i] = 353;
}
else
{
	vR[i] = 352;
}

}

}

}

}

}
else
{
	if(input[i]>(335+offset0))
{
	if(input[i]>(343+offset0))
{
	if(input[i]>(347+offset0))
{
	if(input[i]>(349+offset0))
{
	if(input[i]==(351+offset0))
{
	vR[i] = 351;
}
else
{
	vR[i] = 350;
}

}
else
{
	if(input[i]==(349+offset0))
{
	vR[i] = 349;
}
else
{
	vR[i] = 348;
}

}

}
else
{
	if(input[i]>(345+offset0))
{
	if(input[i]==(347+offset0))
{
	vR[i] = 347;
}
else
{
	vR[i] = 346;
}

}
else
{
	if(input[i]==(345+offset0))
{
	vR[i] = 345;
}
else
{
	vR[i] = 344;
}

}

}

}
else
{
	if(input[i]>(339+offset0))
{
	if(input[i]>(341+offset0))
{
	if(input[i]==(343+offset0))
{
	vR[i] = 343;
}
else
{
	vR[i] = 342;
}

}
else
{
	if(input[i]==(341+offset0))
{
	vR[i] = 341;
}
else
{
	vR[i] = 340;
}

}

}
else
{
	if(input[i]>(337+offset0))
{
	if(input[i]==(339+offset0))
{
	vR[i] = 339;
}
else
{
	vR[i] = 338;
}

}
else
{
	if(input[i]==(337+offset0))
{
	vR[i] = 337;
}
else
{
	vR[i] = 336;
}

}

}

}

}
else
{
	if(input[i]>(327+offset0))
{
	if(input[i]>(331+offset0))
{
	if(input[i]>(333+offset0))
{
	if(input[i]==(335+offset0))
{
	vR[i] = 335;
}
else
{
	vR[i] = 334;
}

}
else
{
	if(input[i]==(333+offset0))
{
	vR[i] = 333;
}
else
{
	vR[i] = 332;
}

}

}
else
{
	if(input[i]>(329+offset0))
{
	if(input[i]==(331+offset0))
{
	vR[i] = 331;
}
else
{
	vR[i] = 330;
}

}
else
{
	if(input[i]==(329+offset0))
{
	vR[i] = 329;
}
else
{
	vR[i] = 328;
}

}

}

}
else
{
	if(input[i]>(323+offset0))
{
	if(input[i]>(325+offset0))
{
	if(input[i]==(327+offset0))
{
	vR[i] = 327;
}
else
{
	vR[i] = 326;
}

}
else
{
	if(input[i]==(325+offset0))
{
	vR[i] = 325;
}
else
{
	vR[i] = 324;
}

}

}
else
{
	if(input[i]>(321+offset0))
{
	if(input[i]==(323+offset0))
{
	vR[i] = 323;
}
else
{
	vR[i] = 322;
}

}
else
{
	if(input[i]==(321+offset0))
{
	vR[i] = 321;
}
else
{
	vR[i] = 320;
}

}

}

}

}

}

}
else
{
	if(input[i]>(287+offset0))
{
	if(input[i]>(303+offset0))
{
	if(input[i]>(311+offset0))
{
	if(input[i]>(315+offset0))
{
	if(input[i]>(317+offset0))
{
	if(input[i]==(319+offset0))
{
	vR[i] = 319;
}
else
{
	vR[i] = 318;
}

}
else
{
	if(input[i]==(317+offset0))
{
	vR[i] = 317;
}
else
{
	vR[i] = 316;
}

}

}
else
{
	if(input[i]>(313+offset0))
{
	if(input[i]==(315+offset0))
{
	vR[i] = 315;
}
else
{
	vR[i] = 314;
}

}
else
{
	if(input[i]==(313+offset0))
{
	vR[i] = 313;
}
else
{
	vR[i] = 312;
}

}

}

}
else
{
	if(input[i]>(307+offset0))
{
	if(input[i]>(309+offset0))
{
	if(input[i]==(311+offset0))
{
	vR[i] = 311;
}
else
{
	vR[i] = 310;
}

}
else
{
	if(input[i]==(309+offset0))
{
	vR[i] = 309;
}
else
{
	vR[i] = 308;
}

}

}
else
{
	if(input[i]>(305+offset0))
{
	if(input[i]==(307+offset0))
{
	vR[i] = 307;
}
else
{
	vR[i] = 306;
}

}
else
{
	if(input[i]==(305+offset0))
{
	vR[i] = 305;
}
else
{
	vR[i] = 304;
}

}

}

}

}
else
{
	if(input[i]>(295+offset0))
{
	if(input[i]>(299+offset0))
{
	if(input[i]>(301+offset0))
{
	if(input[i]==(303+offset0))
{
	vR[i] = 303;
}
else
{
	vR[i] = 302;
}

}
else
{
	if(input[i]==(301+offset0))
{
	vR[i] = 301;
}
else
{
	vR[i] = 300;
}

}

}
else
{
	if(input[i]>(297+offset0))
{
	if(input[i]==(299+offset0))
{
	vR[i] = 299;
}
else
{
	vR[i] = 298;
}

}
else
{
	if(input[i]==(297+offset0))
{
	vR[i] = 297;
}
else
{
	vR[i] = 296;
}

}

}

}
else
{
	if(input[i]>(291+offset0))
{
	if(input[i]>(293+offset0))
{
	if(input[i]==(295+offset0))
{
	vR[i] = 295;
}
else
{
	vR[i] = 294;
}

}
else
{
	if(input[i]==(293+offset0))
{
	vR[i] = 293;
}
else
{
	vR[i] = 292;
}

}

}
else
{
	if(input[i]>(289+offset0))
{
	if(input[i]==(291+offset0))
{
	vR[i] = 291;
}
else
{
	vR[i] = 290;
}

}
else
{
	if(input[i]==(289+offset0))
{
	vR[i] = 289;
}
else
{
	vR[i] = 288;
}

}

}

}

}

}
else
{
	if(input[i]>(271+offset0))
{
	if(input[i]>(279+offset0))
{
	if(input[i]>(283+offset0))
{
	if(input[i]>(285+offset0))
{
	if(input[i]==(287+offset0))
{
	vR[i] = 287;
}
else
{
	vR[i] = 286;
}

}
else
{
	if(input[i]==(285+offset0))
{
	vR[i] = 285;
}
else
{
	vR[i] = 284;
}

}

}
else
{
	if(input[i]>(281+offset0))
{
	if(input[i]==(283+offset0))
{
	vR[i] = 283;
}
else
{
	vR[i] = 282;
}

}
else
{
	if(input[i]==(281+offset0))
{
	vR[i] = 281;
}
else
{
	vR[i] = 280;
}

}

}

}
else
{
	if(input[i]>(275+offset0))
{
	if(input[i]>(277+offset0))
{
	if(input[i]==(279+offset0))
{
	vR[i] = 279;
}
else
{
	vR[i] = 278;
}

}
else
{
	if(input[i]==(277+offset0))
{
	vR[i] = 277;
}
else
{
	vR[i] = 276;
}

}

}
else
{
	if(input[i]>(273+offset0))
{
	if(input[i]==(275+offset0))
{
	vR[i] = 275;
}
else
{
	vR[i] = 274;
}

}
else
{
	if(input[i]==(273+offset0))
{
	vR[i] = 273;
}
else
{
	vR[i] = 272;
}

}

}

}

}
else
{
	if(input[i]>(263+offset0))
{
	if(input[i]>(267+offset0))
{
	if(input[i]>(269+offset0))
{
	if(input[i]==(271+offset0))
{
	vR[i] = 271;
}
else
{
	vR[i] = 270;
}

}
else
{
	if(input[i]==(269+offset0))
{
	vR[i] = 269;
}
else
{
	vR[i] = 268;
}

}

}
else
{
	if(input[i]>(265+offset0))
{
	if(input[i]==(267+offset0))
{
	vR[i] = 267;
}
else
{
	vR[i] = 266;
}

}
else
{
	if(input[i]==(265+offset0))
{
	vR[i] = 265;
}
else
{
	vR[i] = 264;
}

}

}

}
else
{
	if(input[i]>(259+offset0))
{
	if(input[i]>(261+offset0))
{
	if(input[i]==(263+offset0))
{
	vR[i] = 263;
}
else
{
	vR[i] = 262;
}

}
else
{
	if(input[i]==(261+offset0))
{
	vR[i] = 261;
}
else
{
	vR[i] = 260;
}

}

}
else
{
	if(input[i]>(257+offset0))
{
	if(input[i]==(259+offset0))
{
	vR[i] = 259;
}
else
{
	vR[i] = 258;
}

}
else
{
	if(input[i]==(257+offset0))
{
	vR[i] = 257;
}
else
{
	vR[i] = 256;
}

}

}

}

}

}

}

}

}
else
{
	if(input[i]>(127+offset0))
{
	if(input[i]>(191+offset0))
{
	if(input[i]>(223+offset0))
{
	if(input[i]>(239+offset0))
{
	if(input[i]>(247+offset0))
{
	if(input[i]>(251+offset0))
{
	if(input[i]>(253+offset0))
{
	if(input[i]==(255+offset0))
{
	vR[i] = 255;
}
else
{
	vR[i] = 254;
}

}
else
{
	if(input[i]==(253+offset0))
{
	vR[i] = 253;
}
else
{
	vR[i] = 252;
}

}

}
else
{
	if(input[i]>(249+offset0))
{
	if(input[i]==(251+offset0))
{
	vR[i] = 251;
}
else
{
	vR[i] = 250;
}

}
else
{
	if(input[i]==(249+offset0))
{
	vR[i] = 249;
}
else
{
	vR[i] = 248;
}

}

}

}
else
{
	if(input[i]>(243+offset0))
{
	if(input[i]>(245+offset0))
{
	if(input[i]==(247+offset0))
{
	vR[i] = 247;
}
else
{
	vR[i] = 246;
}

}
else
{
	if(input[i]==(245+offset0))
{
	vR[i] = 245;
}
else
{
	vR[i] = 244;
}

}

}
else
{
	if(input[i]>(241+offset0))
{
	if(input[i]==(243+offset0))
{
	vR[i] = 243;
}
else
{
	vR[i] = 242;
}

}
else
{
	if(input[i]==(241+offset0))
{
	vR[i] = 241;
}
else
{
	vR[i] = 240;
}

}

}

}

}
else
{
	if(input[i]>(231+offset0))
{
	if(input[i]>(235+offset0))
{
	if(input[i]>(237+offset0))
{
	if(input[i]==(239+offset0))
{
	vR[i] = 239;
}
else
{
	vR[i] = 238;
}

}
else
{
	if(input[i]==(237+offset0))
{
	vR[i] = 237;
}
else
{
	vR[i] = 236;
}

}

}
else
{
	if(input[i]>(233+offset0))
{
	if(input[i]==(235+offset0))
{
	vR[i] = 235;
}
else
{
	vR[i] = 234;
}

}
else
{
	if(input[i]==(233+offset0))
{
	vR[i] = 233;
}
else
{
	vR[i] = 232;
}

}

}

}
else
{
	if(input[i]>(227+offset0))
{
	if(input[i]>(229+offset0))
{
	if(input[i]==(231+offset0))
{
	vR[i] = 231;
}
else
{
	vR[i] = 230;
}

}
else
{
	if(input[i]==(229+offset0))
{
	vR[i] = 229;
}
else
{
	vR[i] = 228;
}

}

}
else
{
	if(input[i]>(225+offset0))
{
	if(input[i]==(227+offset0))
{
	vR[i] = 227;
}
else
{
	vR[i] = 226;
}

}
else
{
	if(input[i]==(225+offset0))
{
	vR[i] = 225;
}
else
{
	vR[i] = 224;
}

}

}

}

}

}
else
{
	if(input[i]>(207+offset0))
{
	if(input[i]>(215+offset0))
{
	if(input[i]>(219+offset0))
{
	if(input[i]>(221+offset0))
{
	if(input[i]==(223+offset0))
{
	vR[i] = 223;
}
else
{
	vR[i] = 222;
}

}
else
{
	if(input[i]==(221+offset0))
{
	vR[i] = 221;
}
else
{
	vR[i] = 220;
}

}

}
else
{
	if(input[i]>(217+offset0))
{
	if(input[i]==(219+offset0))
{
	vR[i] = 219;
}
else
{
	vR[i] = 218;
}

}
else
{
	if(input[i]==(217+offset0))
{
	vR[i] = 217;
}
else
{
	vR[i] = 216;
}

}

}

}
else
{
	if(input[i]>(211+offset0))
{
	if(input[i]>(213+offset0))
{
	if(input[i]==(215+offset0))
{
	vR[i] = 215;
}
else
{
	vR[i] = 214;
}

}
else
{
	if(input[i]==(213+offset0))
{
	vR[i] = 213;
}
else
{
	vR[i] = 212;
}

}

}
else
{
	if(input[i]>(209+offset0))
{
	if(input[i]==(211+offset0))
{
	vR[i] = 211;
}
else
{
	vR[i] = 210;
}

}
else
{
	if(input[i]==(209+offset0))
{
	vR[i] = 209;
}
else
{
	vR[i] = 208;
}

}

}

}

}
else
{
	if(input[i]>(199+offset0))
{
	if(input[i]>(203+offset0))
{
	if(input[i]>(205+offset0))
{
	if(input[i]==(207+offset0))
{
	vR[i] = 207;
}
else
{
	vR[i] = 206;
}

}
else
{
	if(input[i]==(205+offset0))
{
	vR[i] = 205;
}
else
{
	vR[i] = 204;
}

}

}
else
{
	if(input[i]>(201+offset0))
{
	if(input[i]==(203+offset0))
{
	vR[i] = 203;
}
else
{
	vR[i] = 202;
}

}
else
{
	if(input[i]==(201+offset0))
{
	vR[i] = 201;
}
else
{
	vR[i] = 200;
}

}

}

}
else
{
	if(input[i]>(195+offset0))
{
	if(input[i]>(197+offset0))
{
	if(input[i]==(199+offset0))
{
	vR[i] = 199;
}
else
{
	vR[i] = 198;
}

}
else
{
	if(input[i]==(197+offset0))
{
	vR[i] = 197;
}
else
{
	vR[i] = 196;
}

}

}
else
{
	if(input[i]>(193+offset0))
{
	if(input[i]==(195+offset0))
{
	vR[i] = 195;
}
else
{
	vR[i] = 194;
}

}
else
{
	if(input[i]==(193+offset0))
{
	vR[i] = 193;
}
else
{
	vR[i] = 192;
}

}

}

}

}

}

}
else
{
	if(input[i]>(159+offset0))
{
	if(input[i]>(175+offset0))
{
	if(input[i]>(183+offset0))
{
	if(input[i]>(187+offset0))
{
	if(input[i]>(189+offset0))
{
	if(input[i]==(191+offset0))
{
	vR[i] = 191;
}
else
{
	vR[i] = 190;
}

}
else
{
	if(input[i]==(189+offset0))
{
	vR[i] = 189;
}
else
{
	vR[i] = 188;
}

}

}
else
{
	if(input[i]>(185+offset0))
{
	if(input[i]==(187+offset0))
{
	vR[i] = 187;
}
else
{
	vR[i] = 186;
}

}
else
{
	if(input[i]==(185+offset0))
{
	vR[i] = 185;
}
else
{
	vR[i] = 184;
}

}

}

}
else
{
	if(input[i]>(179+offset0))
{
	if(input[i]>(181+offset0))
{
	if(input[i]==(183+offset0))
{
	vR[i] = 183;
}
else
{
	vR[i] = 182;
}

}
else
{
	if(input[i]==(181+offset0))
{
	vR[i] = 181;
}
else
{
	vR[i] = 180;
}

}

}
else
{
	if(input[i]>(177+offset0))
{
	if(input[i]==(179+offset0))
{
	vR[i] = 179;
}
else
{
	vR[i] = 178;
}

}
else
{
	if(input[i]==(177+offset0))
{
	vR[i] = 177;
}
else
{
	vR[i] = 176;
}

}

}

}

}
else
{
	if(input[i]>(167+offset0))
{
	if(input[i]>(171+offset0))
{
	if(input[i]>(173+offset0))
{
	if(input[i]==(175+offset0))
{
	vR[i] = 175;
}
else
{
	vR[i] = 174;
}

}
else
{
	if(input[i]==(173+offset0))
{
	vR[i] = 173;
}
else
{
	vR[i] = 172;
}

}

}
else
{
	if(input[i]>(169+offset0))
{
	if(input[i]==(171+offset0))
{
	vR[i] = 171;
}
else
{
	vR[i] = 170;
}

}
else
{
	if(input[i]==(169+offset0))
{
	vR[i] = 169;
}
else
{
	vR[i] = 168;
}

}

}

}
else
{
	if(input[i]>(163+offset0))
{
	if(input[i]>(165+offset0))
{
	if(input[i]==(167+offset0))
{
	vR[i] = 167;
}
else
{
	vR[i] = 166;
}

}
else
{
	if(input[i]==(165+offset0))
{
	vR[i] = 165;
}
else
{
	vR[i] = 164;
}

}

}
else
{
	if(input[i]>(161+offset0))
{
	if(input[i]==(163+offset0))
{
	vR[i] = 163;
}
else
{
	vR[i] = 162;
}

}
else
{
	if(input[i]==(161+offset0))
{
	vR[i] = 161;
}
else
{
	vR[i] = 160;
}

}

}

}

}

}
else
{
	if(input[i]>(143+offset0))
{
	if(input[i]>(151+offset0))
{
	if(input[i]>(155+offset0))
{
	if(input[i]>(157+offset0))
{
	if(input[i]==(159+offset0))
{
	vR[i] = 159;
}
else
{
	vR[i] = 158;
}

}
else
{
	if(input[i]==(157+offset0))
{
	vR[i] = 157;
}
else
{
	vR[i] = 156;
}

}

}
else
{
	if(input[i]>(153+offset0))
{
	if(input[i]==(155+offset0))
{
	vR[i] = 155;
}
else
{
	vR[i] = 154;
}

}
else
{
	if(input[i]==(153+offset0))
{
	vR[i] = 153;
}
else
{
	vR[i] = 152;
}

}

}

}
else
{
	if(input[i]>(147+offset0))
{
	if(input[i]>(149+offset0))
{
	if(input[i]==(151+offset0))
{
	vR[i] = 151;
}
else
{
	vR[i] = 150;
}

}
else
{
	if(input[i]==(149+offset0))
{
	vR[i] = 149;
}
else
{
	vR[i] = 148;
}

}

}
else
{
	if(input[i]>(145+offset0))
{
	if(input[i]==(147+offset0))
{
	vR[i] = 147;
}
else
{
	vR[i] = 146;
}

}
else
{
	if(input[i]==(145+offset0))
{
	vR[i] = 145;
}
else
{
	vR[i] = 144;
}

}

}

}

}
else
{
	if(input[i]>(135+offset0))
{
	if(input[i]>(139+offset0))
{
	if(input[i]>(141+offset0))
{
	if(input[i]==(143+offset0))
{
	vR[i] = 143;
}
else
{
	vR[i] = 142;
}

}
else
{
	if(input[i]==(141+offset0))
{
	vR[i] = 141;
}
else
{
	vR[i] = 140;
}

}

}
else
{
	if(input[i]>(137+offset0))
{
	if(input[i]==(139+offset0))
{
	vR[i] = 139;
}
else
{
	vR[i] = 138;
}

}
else
{
	if(input[i]==(137+offset0))
{
	vR[i] = 137;
}
else
{
	vR[i] = 136;
}

}

}

}
else
{
	if(input[i]>(131+offset0))
{
	if(input[i]>(133+offset0))
{
	if(input[i]==(135+offset0))
{
	vR[i] = 135;
}
else
{
	vR[i] = 134;
}

}
else
{
	if(input[i]==(133+offset0))
{
	vR[i] = 133;
}
else
{
	vR[i] = 132;
}

}

}
else
{
	if(input[i]>(129+offset0))
{
	if(input[i]==(131+offset0))
{
	vR[i] = 131;
}
else
{
	vR[i] = 130;
}

}
else
{
	if(input[i]==(129+offset0))
{
	vR[i] = 129;
}
else
{
	vR[i] = 128;
}

}

}

}

}

}

}

}
else
{
	if(input[i]>(63+offset0))
{
	if(input[i]>(95+offset0))
{
	if(input[i]>(111+offset0))
{
	if(input[i]>(119+offset0))
{
	if(input[i]>(123+offset0))
{
	if(input[i]>(125+offset0))
{
	if(input[i]==(127+offset0))
{
	vR[i] = 127;
}
else
{
	vR[i] = 126;
}

}
else
{
	if(input[i]==(125+offset0))
{
	vR[i] = 125;
}
else
{
	vR[i] = 124;
}

}

}
else
{
	if(input[i]>(121+offset0))
{
	if(input[i]==(123+offset0))
{
	vR[i] = 123;
}
else
{
	vR[i] = 122;
}

}
else
{
	if(input[i]==(121+offset0))
{
	vR[i] = 121;
}
else
{
	vR[i] = 120;
}

}

}

}
else
{
	if(input[i]>(115+offset0))
{
	if(input[i]>(117+offset0))
{
	if(input[i]==(119+offset0))
{
	vR[i] = 119;
}
else
{
	vR[i] = 118;
}

}
else
{
	if(input[i]==(117+offset0))
{
	vR[i] = 117;
}
else
{
	vR[i] = 116;
}

}

}
else
{
	if(input[i]>(113+offset0))
{
	if(input[i]==(115+offset0))
{
	vR[i] = 115;
}
else
{
	vR[i] = 114;
}

}
else
{
	if(input[i]==(113+offset0))
{
	vR[i] = 113;
}
else
{
	vR[i] = 112;
}

}

}

}

}
else
{
	if(input[i]>(103+offset0))
{
	if(input[i]>(107+offset0))
{
	if(input[i]>(109+offset0))
{
	if(input[i]==(111+offset0))
{
	vR[i] = 111;
}
else
{
	vR[i] = 110;
}

}
else
{
	if(input[i]==(109+offset0))
{
	vR[i] = 109;
}
else
{
	vR[i] = 108;
}

}

}
else
{
	if(input[i]>(105+offset0))
{
	if(input[i]==(107+offset0))
{
	vR[i] = 107;
}
else
{
	vR[i] = 106;
}

}
else
{
	if(input[i]==(105+offset0))
{
	vR[i] = 105;
}
else
{
	vR[i] = 104;
}

}

}

}
else
{
	if(input[i]>(99+offset0))
{
	if(input[i]>(101+offset0))
{
	if(input[i]==(103+offset0))
{
	vR[i] = 103;
}
else
{
	vR[i] = 102;
}

}
else
{
	if(input[i]==(101+offset0))
{
	vR[i] = 101;
}
else
{
	vR[i] = 100;
}

}

}
else
{
	if(input[i]>(97+offset0))
{
	if(input[i]==(99+offset0))
{
	vR[i] = 99;
}
else
{
	vR[i] = 98;
}

}
else
{
	if(input[i]==(97+offset0))
{
	vR[i] = 97;
}
else
{
	vR[i] = 96;
}

}

}

}

}

}
else
{
	if(input[i]>(79+offset0))
{
	if(input[i]>(87+offset0))
{
	if(input[i]>(91+offset0))
{
	if(input[i]>(93+offset0))
{
	if(input[i]==(95+offset0))
{
	vR[i] = 95;
}
else
{
	vR[i] = 94;
}

}
else
{
	if(input[i]==(93+offset0))
{
	vR[i] = 93;
}
else
{
	vR[i] = 92;
}

}

}
else
{
	if(input[i]>(89+offset0))
{
	if(input[i]==(91+offset0))
{
	vR[i] = 91;
}
else
{
	vR[i] = 90;
}

}
else
{
	if(input[i]==(89+offset0))
{
	vR[i] = 89;
}
else
{
	vR[i] = 88;
}

}

}

}
else
{
	if(input[i]>(83+offset0))
{
	if(input[i]>(85+offset0))
{
	if(input[i]==(87+offset0))
{
	vR[i] = 87;
}
else
{
	vR[i] = 86;
}

}
else
{
	if(input[i]==(85+offset0))
{
	vR[i] = 85;
}
else
{
	vR[i] = 84;
}

}

}
else
{
	if(input[i]>(81+offset0))
{
	if(input[i]==(83+offset0))
{
	vR[i] = 83;
}
else
{
	vR[i] = 82;
}

}
else
{
	if(input[i]==(81+offset0))
{
	vR[i] = 81;
}
else
{
	vR[i] = 80;
}

}

}

}

}
else
{
	if(input[i]>(71+offset0))
{
	if(input[i]>(75+offset0))
{
	if(input[i]>(77+offset0))
{
	if(input[i]==(79+offset0))
{
	vR[i] = 79;
}
else
{
	vR[i] = 78;
}

}
else
{
	if(input[i]==(77+offset0))
{
	vR[i] = 77;
}
else
{
	vR[i] = 76;
}

}

}
else
{
	if(input[i]>(73+offset0))
{
	if(input[i]==(75+offset0))
{
	vR[i] = 75;
}
else
{
	vR[i] = 74;
}

}
else
{
	if(input[i]==(73+offset0))
{
	vR[i] = 73;
}
else
{
	vR[i] = 72;
}

}

}

}
else
{
	if(input[i]>(67+offset0))
{
	if(input[i]>(69+offset0))
{
	if(input[i]==(71+offset0))
{
	vR[i] = 71;
}
else
{
	vR[i] = 70;
}

}
else
{
	if(input[i]==(69+offset0))
{
	vR[i] = 69;
}
else
{
	vR[i] = 68;
}

}

}
else
{
	if(input[i]>(65+offset0))
{
	if(input[i]==(67+offset0))
{
	vR[i] = 67;
}
else
{
	vR[i] = 66;
}

}
else
{
	if(input[i]==(65+offset0))
{
	vR[i] = 65;
}
else
{
	vR[i] = 64;
}

}

}

}

}

}

}
else
{
	if(input[i]>(31+offset0))
{
	if(input[i]>(47+offset0))
{
	if(input[i]>(55+offset0))
{
	if(input[i]>(59+offset0))
{
	if(input[i]>(61+offset0))
{
	if(input[i]==(63+offset0))
{
	vR[i] = 63;
}
else
{
	vR[i] = 62;
}

}
else
{
	if(input[i]==(61+offset0))
{
	vR[i] = 61;
}
else
{
	vR[i] = 60;
}

}

}
else
{
	if(input[i]>(57+offset0))
{
	if(input[i]==(59+offset0))
{
	vR[i] = 59;
}
else
{
	vR[i] = 58;
}

}
else
{
	if(input[i]==(57+offset0))
{
	vR[i] = 57;
}
else
{
	vR[i] = 56;
}

}

}

}
else
{
	if(input[i]>(51+offset0))
{
	if(input[i]>(53+offset0))
{
	if(input[i]==(55+offset0))
{
	vR[i] = 55;
}
else
{
	vR[i] = 54;
}

}
else
{
	if(input[i]==(53+offset0))
{
	vR[i] = 53;
}
else
{
	vR[i] = 52;
}

}

}
else
{
	if(input[i]>(49+offset0))
{
	if(input[i]==(51+offset0))
{
	vR[i] = 51;
}
else
{
	vR[i] = 50;
}

}
else
{
	if(input[i]==(49+offset0))
{
	vR[i] = 49;
}
else
{
	vR[i] = 48;
}

}

}

}

}
else
{
	if(input[i]>(39+offset0))
{
	if(input[i]>(43+offset0))
{
	if(input[i]>(45+offset0))
{
	if(input[i]==(47+offset0))
{
	vR[i] = 47;
}
else
{
	vR[i] = 46;
}

}
else
{
	if(input[i]==(45+offset0))
{
	vR[i] = 45;
}
else
{
	vR[i] = 44;
}

}

}
else
{
	if(input[i]>(41+offset0))
{
	if(input[i]==(43+offset0))
{
	vR[i] = 43;
}
else
{
	vR[i] = 42;
}

}
else
{
	if(input[i]==(41+offset0))
{
	vR[i] = 41;
}
else
{
	vR[i] = 40;
}

}

}

}
else
{
	if(input[i]>(35+offset0))
{
	if(input[i]>(37+offset0))
{
	if(input[i]==(39+offset0))
{
	vR[i] = 39;
}
else
{
	vR[i] = 38;
}

}
else
{
	if(input[i]==(37+offset0))
{
	vR[i] = 37;
}
else
{
	vR[i] = 36;
}

}

}
else
{
	if(input[i]>(33+offset0))
{
	if(input[i]==(35+offset0))
{
	vR[i] = 35;
}
else
{
	vR[i] = 34;
}

}
else
{
	if(input[i]==(33+offset0))
{
	vR[i] = 33;
}
else
{
	vR[i] = 32;
}

}

}

}

}

}
else
{
	if(input[i]>(15+offset0))
{
	if(input[i]>(23+offset0))
{
	if(input[i]>(27+offset0))
{
	if(input[i]>(29+offset0))
{
	if(input[i]==(31+offset0))
{
	vR[i] = 31;
}
else
{
	vR[i] = 30;
}

}
else
{
	if(input[i]==(29+offset0))
{
	vR[i] = 29;
}
else
{
	vR[i] = 28;
}

}

}
else
{
	if(input[i]>(25+offset0))
{
	if(input[i]==(27+offset0))
{
	vR[i] = 27;
}
else
{
	vR[i] = 26;
}

}
else
{
	if(input[i]==(25+offset0))
{
	vR[i] = 25;
}
else
{
	vR[i] = 24;
}

}

}

}
else
{
	if(input[i]>(19+offset0))
{
	if(input[i]>(21+offset0))
{
	if(input[i]==(23+offset0))
{
	vR[i] = 23;
}
else
{
	vR[i] = 22;
}

}
else
{
	if(input[i]==(21+offset0))
{
	vR[i] = 21;
}
else
{
	vR[i] = 20;
}

}

}
else
{
	if(input[i]>(17+offset0))
{
	if(input[i]==(19+offset0))
{
	vR[i] = 19;
}
else
{
	vR[i] = 18;
}

}
else
{
	if(input[i]==(17+offset0))
{
	vR[i] = 17;
}
else
{
	vR[i] = 16;
}

}

}

}

}
else
{
	if(input[i]>(7+offset0))
{
	if(input[i]>(11+offset0))
{
	if(input[i]>(13+offset0))
{
	if(input[i]==(15+offset0))
{
	vR[i] = 15;
}
else
{
	vR[i] = 14;
}

}
else
{
	if(input[i]==(13+offset0))
{
	vR[i] = 13;
}
else
{
	vR[i] = 12;
}

}

}
else
{
	if(input[i]>(9+offset0))
{
	if(input[i]==(11+offset0))
{
	vR[i] = 11;
}
else
{
	vR[i] = 10;
}

}
else
{
	if(input[i]==(9+offset0))
{
	vR[i] = 9;
}
else
{
	vR[i] = 8;
}

}

}

}
else
{
	if(input[i]>(3+offset0))
{
	if(input[i]>(5+offset0))
{
	if(input[i]==(7+offset0))
{
	vR[i] = 7;
}
else
{
	vR[i] = 6;
}

}
else
{
	if(input[i]==(5+offset0))
{
	vR[i] = 5;
}
else
{
	vR[i] = 4;
}

}

}
else
{
	if(input[i]>(1+offset0))
{
	if(input[i]==(3+offset0))
{
	vR[i] = 3;
}
else
{
	vR[i] = 2;
}

}
else
{
	if(input[i]==(1+offset0))
{
	vR[i] = 1;
}
else
{
	vR[i] = 0;
}

}

}

}

}

}

}

}

}

}

}

}


	}
}

int main(int argc, char **argv)
{
	float elapsed_time[TIMES], avg = 0.0, sd = 0.0;
	// set up device
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("[CUDA - Random Forest]: EXP %s\n", EXP_NAME);
	printf("[CUDA - Random Forest]: Using Device %d: %s\n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));

	// set up data size of vectors
	int nElem = N_ELEM;
	printf("[CUDA - Random Forest]: Vector Size %d\n", nElem);

	// malloc host memory
	size_t nBytes = nElem * sizeof(float);
	float *h_vR;
	h_vR = (float *)malloc(nBytes);
	float *h_input;
	h_input = (float *)malloc(nBytes);

	printf("[CUDA - Random Forest]: Start Reading Dataset\n");
	initData(h_input, nElem);
	printf("[CUDA - Random Forest]: Finished Reading Dataset\n");
	memset(h_vR, 0, nBytes);

	// malloc device global memory
	float *d_vR;
	CHECK(hipMalloc((float **)&d_vR, nBytes));
	float *d_input;
	CHECK(hipMalloc((float **)&d_input, nBytes));

	// transfer data from host to device
	CHECK(hipMemcpy(d_input, h_input, nBytes, hipMemcpyHostToDevice));

	// invoke kernel at host side
	int iLen = 512;
	dim3 block(iLen);
	dim3 grid((nElem + block.x - 1) / block.x);
	hipEvent_t start, stop;
	CHECK(hipEventCreate(&start));
	CHECK(hipEventCreate(&stop));

	for (int t = 0; t < TIMES; t++)
	{
	// record start event
		CHECK(hipEventRecord(start, 0));
#if defined(_2TREE)
		RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, d_vR);
#elif defined(_3TREE)
		RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, d_vR);
#elif defined(_4TREE)
		RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, d_vR);
#elif defined(_5TREE)
		RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, 0, d_vR);
#elif defined(_6TREE)
		RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, 0, 0, d_vR);
#elif defined(_7TREE)
		RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, 0, 0, 0, d_vR);
#elif defined(_8TREE)
		RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, 0, 0, 0, 0, d_vR);
#elif defined(_9TREE)
		RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, 0, 0, 0, 0, 0, d_vR);
#elif defined(_10TREE)
		RF_kernel<<<grid, block>>>(d_input, nElem, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, d_vR);
#else
		RF_kernel<<<grid, block>>>(d_input, nElem, 0, d_vR);
#endif
		CHECK(hipEventRecord(stop, 0));
		CHECK(hipEventSynchronize(stop));
		hipDeviceSynchronize();

		// calculate elapsed time
		CHECK(hipEventElapsedTime(&elapsed_time[t], start, stop));
		printf("[CUDA - Random Forest]: %s - execution time = %.6fms\n", EXP_NAME, elapsed_time[t]);
		registerTime(elapsed_time[t]);

		avg += elapsed_time[t];
	}
	// copy kernel result back to host side
	CHECK(hipMemcpy(h_vR, d_vR, nBytes, hipMemcpyDeviceToHost));
	CHECK(hipGetLastError());

	//Verify the answers
	char flag = 1;
	for (int i = 0; i < nElem; i++)
	{
		if (i < 32)
		{
			//printf("%.3f ", h_vR[i]);
		}
		if (h_input[i] != h_vR[i])
		{
			flag = 0;
			break;
		}
	}
	if (flag == 1)
	{
		printf("Return correct");
	}
	else
	{
		printf("Error!!");
	}
	printf("\n ");

	// avg and standard deviation
	avg = (float)avg / TIMES;
	for (int t = 0; t < TIMES; t++)
	{
		sd+=pow(elapsed_time[t]-avg,2);
	}
	sd = (float) sd/TIMES;
	sd = sqrt(sd);
	printf("\nAVG = %fms, SD = %fms\n", avg,sd);
	// free device memory
	hipFree(d_input);
	hipFree(d_vR);

	// free host memory
	free(h_input);
	free(h_vR);
	return (0);
}

void initData(float *ip, int size)
{
	for (int i = 0; i < size; i++)
	{

#if defined(_BEST)
		ip[i] = (float)(0);
#else
		if(WARP_SIZE < _MOD)
		{
			int aux = _MOD / WARP_SIZE;
			ip[i] = (float)((i * aux) % _MOD);
		}
		else
		{
			ip[i] = (float)(i % _MOD);
		}
#endif
	}
	return;
}

void registerTime(float value)
{
	printf("[CUDA - Random Forest]: Time spent: %f.\n", value);
}
